#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ NondetRegStruct exec_IsZero(ExecContext& ctx,
                                       Val arg0,
                                       BoundLayout<IsZeroLayout> layout1) {
  // IsZero(zirgen/circuit/keccak2/is_zero.zir:8)
  NondetRegStruct x2 = exec_NondetReg(ctx, isz(arg0), LAYOUT_LOOKUP(layout1, _super));
  // IsZero(zirgen/circuit/keccak2/is_zero.zir:11)
  NondetRegStruct x3 = exec_NondetReg(ctx, inv_0(arg0), LAYOUT_LOOKUP(layout1, inv));
  // AssertBit(zirgen/circuit/keccak2/bits.zir:6)
  // IsZero(zirgen/circuit/keccak2/is_zero.zir:14)
  Val x4 = (Val(1) - x2._super);
  EQZ((x2._super * x4),
      "loc(callsite( AssertBit ( zirgen/circuit/keccak2/bits.zir :6:20) at  IsZero ( "
      "zirgen/circuit/keccak2/is_zero.zir :14:13)))");
  // IsZero(zirgen/circuit/keccak2/is_zero.zir:16)
  EQZ(((arg0 * x3._super) - x4), "IsZero(zirgen/circuit/keccak2/is_zero.zir:16)");
  // IsZero(zirgen/circuit/keccak2/is_zero.zir:18)
  EQZ((x2._super * arg0), "IsZero(zirgen/circuit/keccak2/is_zero.zir:18)");
  // IsZero(zirgen/circuit/keccak2/is_zero.zir:20)
  EQZ((x2._super * x3._super), "IsZero(zirgen/circuit/keccak2/is_zero.zir:20)");
  return x2;
}
__device__ TopStateStruct exec_ShaCycle(ExecContext& ctx,
                                        TopStateStruct arg0,
                                        TopStateStruct arg1,
                                        Val arg2,
                                        Val arg3,
                                        BoundLayout<ShaCycleLayout> layout4) {
  // Log(<preamble>:22)
  // ShaCycle(zirgen/circuit/keccak2/top.zir:187)
  INVOKE_EXTERN(ctx, log, "ShaCycle", std::initializer_list<Val>{arg3, arg2});
  // LoadShaState(zirgen/circuit/keccak2/top.zir:148)
  // ShaCycle(zirgen/circuit/keccak2/top.zir:189)
  Val32Array x5 = Val32Array{
      arg0.bits[224]._super._super, arg0.bits[225]._super._super, arg0.bits[226]._super._super,
      arg0.bits[227]._super._super, arg0.bits[228]._super._super, arg0.bits[229]._super._super,
      arg0.bits[230]._super._super, arg0.bits[231]._super._super, arg0.bits[232]._super._super,
      arg0.bits[233]._super._super, arg0.bits[234]._super._super, arg0.bits[235]._super._super,
      arg0.bits[236]._super._super, arg0.bits[237]._super._super, arg0.bits[238]._super._super,
      arg0.bits[239]._super._super, arg0.bits[240]._super._super, arg0.bits[241]._super._super,
      arg0.bits[242]._super._super, arg0.bits[243]._super._super, arg0.bits[244]._super._super,
      arg0.bits[245]._super._super, arg0.bits[246]._super._super, arg0.bits[247]._super._super,
      arg0.bits[248]._super._super, arg0.bits[249]._super._super, arg0.bits[250]._super._super,
      arg0.bits[251]._super._super, arg0.bits[252]._super._super, arg0.bits[253]._super._super,
      arg0.bits[254]._super._super, arg0.bits[255]._super._super};
  Val32Array x6 = Val32Array{
      arg0.bits[192]._super._super, arg0.bits[193]._super._super, arg0.bits[194]._super._super,
      arg0.bits[195]._super._super, arg0.bits[196]._super._super, arg0.bits[197]._super._super,
      arg0.bits[198]._super._super, arg0.bits[199]._super._super, arg0.bits[200]._super._super,
      arg0.bits[201]._super._super, arg0.bits[202]._super._super, arg0.bits[203]._super._super,
      arg0.bits[204]._super._super, arg0.bits[205]._super._super, arg0.bits[206]._super._super,
      arg0.bits[207]._super._super, arg0.bits[208]._super._super, arg0.bits[209]._super._super,
      arg0.bits[210]._super._super, arg0.bits[211]._super._super, arg0.bits[212]._super._super,
      arg0.bits[213]._super._super, arg0.bits[214]._super._super, arg0.bits[215]._super._super,
      arg0.bits[216]._super._super, arg0.bits[217]._super._super, arg0.bits[218]._super._super,
      arg0.bits[219]._super._super, arg0.bits[220]._super._super, arg0.bits[221]._super._super,
      arg0.bits[222]._super._super, arg0.bits[223]._super._super};
  Val32Array x7 = Val32Array{
      arg0.bits[160]._super._super, arg0.bits[161]._super._super, arg0.bits[162]._super._super,
      arg0.bits[163]._super._super, arg0.bits[164]._super._super, arg0.bits[165]._super._super,
      arg0.bits[166]._super._super, arg0.bits[167]._super._super, arg0.bits[168]._super._super,
      arg0.bits[169]._super._super, arg0.bits[170]._super._super, arg0.bits[171]._super._super,
      arg0.bits[172]._super._super, arg0.bits[173]._super._super, arg0.bits[174]._super._super,
      arg0.bits[175]._super._super, arg0.bits[176]._super._super, arg0.bits[177]._super._super,
      arg0.bits[178]._super._super, arg0.bits[179]._super._super, arg0.bits[180]._super._super,
      arg0.bits[181]._super._super, arg0.bits[182]._super._super, arg0.bits[183]._super._super,
      arg0.bits[184]._super._super, arg0.bits[185]._super._super, arg0.bits[186]._super._super,
      arg0.bits[187]._super._super, arg0.bits[188]._super._super, arg0.bits[189]._super._super,
      arg0.bits[190]._super._super, arg0.bits[191]._super._super};
  Val32Array x8 = Val32Array{
      arg0.bits[128]._super._super, arg0.bits[129]._super._super, arg0.bits[130]._super._super,
      arg0.bits[131]._super._super, arg0.bits[132]._super._super, arg0.bits[133]._super._super,
      arg0.bits[134]._super._super, arg0.bits[135]._super._super, arg0.bits[136]._super._super,
      arg0.bits[137]._super._super, arg0.bits[138]._super._super, arg0.bits[139]._super._super,
      arg0.bits[140]._super._super, arg0.bits[141]._super._super, arg0.bits[142]._super._super,
      arg0.bits[143]._super._super, arg0.bits[144]._super._super, arg0.bits[145]._super._super,
      arg0.bits[146]._super._super, arg0.bits[147]._super._super, arg0.bits[148]._super._super,
      arg0.bits[149]._super._super, arg0.bits[150]._super._super, arg0.bits[151]._super._super,
      arg0.bits[152]._super._super, arg0.bits[153]._super._super, arg0.bits[154]._super._super,
      arg0.bits[155]._super._super, arg0.bits[156]._super._super, arg0.bits[157]._super._super,
      arg0.bits[158]._super._super, arg0.bits[159]._super._super};
  // LoadShaState(zirgen/circuit/keccak2/top.zir:149)
  Val32Array x9 = Val32Array{
      arg0.bits[480]._super._super, arg0.bits[481]._super._super, arg0.bits[482]._super._super,
      arg0.bits[483]._super._super, arg0.bits[484]._super._super, arg0.bits[485]._super._super,
      arg0.bits[486]._super._super, arg0.bits[487]._super._super, arg0.bits[488]._super._super,
      arg0.bits[489]._super._super, arg0.bits[490]._super._super, arg0.bits[491]._super._super,
      arg0.bits[492]._super._super, arg0.bits[493]._super._super, arg0.bits[494]._super._super,
      arg0.bits[495]._super._super, arg0.bits[496]._super._super, arg0.bits[497]._super._super,
      arg0.bits[498]._super._super, arg0.bits[499]._super._super, arg0.bits[500]._super._super,
      arg0.bits[501]._super._super, arg0.bits[502]._super._super, arg0.bits[503]._super._super,
      arg0.bits[504]._super._super, arg0.bits[505]._super._super, arg0.bits[506]._super._super,
      arg0.bits[507]._super._super, arg0.bits[508]._super._super, arg0.bits[509]._super._super,
      arg0.bits[510]._super._super, arg0.bits[511]._super._super};
  Val32Array x10 = Val32Array{
      arg0.bits[448]._super._super, arg0.bits[449]._super._super, arg0.bits[450]._super._super,
      arg0.bits[451]._super._super, arg0.bits[452]._super._super, arg0.bits[453]._super._super,
      arg0.bits[454]._super._super, arg0.bits[455]._super._super, arg0.bits[456]._super._super,
      arg0.bits[457]._super._super, arg0.bits[458]._super._super, arg0.bits[459]._super._super,
      arg0.bits[460]._super._super, arg0.bits[461]._super._super, arg0.bits[462]._super._super,
      arg0.bits[463]._super._super, arg0.bits[464]._super._super, arg0.bits[465]._super._super,
      arg0.bits[466]._super._super, arg0.bits[467]._super._super, arg0.bits[468]._super._super,
      arg0.bits[469]._super._super, arg0.bits[470]._super._super, arg0.bits[471]._super._super,
      arg0.bits[472]._super._super, arg0.bits[473]._super._super, arg0.bits[474]._super._super,
      arg0.bits[475]._super._super, arg0.bits[476]._super._super, arg0.bits[477]._super._super,
      arg0.bits[478]._super._super, arg0.bits[479]._super._super};
  Val32Array x11 = Val32Array{
      arg0.bits[416]._super._super, arg0.bits[417]._super._super, arg0.bits[418]._super._super,
      arg0.bits[419]._super._super, arg0.bits[420]._super._super, arg0.bits[421]._super._super,
      arg0.bits[422]._super._super, arg0.bits[423]._super._super, arg0.bits[424]._super._super,
      arg0.bits[425]._super._super, arg0.bits[426]._super._super, arg0.bits[427]._super._super,
      arg0.bits[428]._super._super, arg0.bits[429]._super._super, arg0.bits[430]._super._super,
      arg0.bits[431]._super._super, arg0.bits[432]._super._super, arg0.bits[433]._super._super,
      arg0.bits[434]._super._super, arg0.bits[435]._super._super, arg0.bits[436]._super._super,
      arg0.bits[437]._super._super, arg0.bits[438]._super._super, arg0.bits[439]._super._super,
      arg0.bits[440]._super._super, arg0.bits[441]._super._super, arg0.bits[442]._super._super,
      arg0.bits[443]._super._super, arg0.bits[444]._super._super, arg0.bits[445]._super._super,
      arg0.bits[446]._super._super, arg0.bits[447]._super._super};
  Val32Array x12 = Val32Array{
      arg0.bits[384]._super._super, arg0.bits[385]._super._super, arg0.bits[386]._super._super,
      arg0.bits[387]._super._super, arg0.bits[388]._super._super, arg0.bits[389]._super._super,
      arg0.bits[390]._super._super, arg0.bits[391]._super._super, arg0.bits[392]._super._super,
      arg0.bits[393]._super._super, arg0.bits[394]._super._super, arg0.bits[395]._super._super,
      arg0.bits[396]._super._super, arg0.bits[397]._super._super, arg0.bits[398]._super._super,
      arg0.bits[399]._super._super, arg0.bits[400]._super._super, arg0.bits[401]._super._super,
      arg0.bits[402]._super._super, arg0.bits[403]._super._super, arg0.bits[404]._super._super,
      arg0.bits[405]._super._super, arg0.bits[406]._super._super, arg0.bits[407]._super._super,
      arg0.bits[408]._super._super, arg0.bits[409]._super._super, arg0.bits[410]._super._super,
      arg0.bits[411]._super._super, arg0.bits[412]._super._super, arg0.bits[413]._super._super,
      arg0.bits[414]._super._super, arg0.bits[415]._super._super};
  // LoadShaState(zirgen/circuit/keccak2/top.zir:150)
  Val32Array x13 = Val32Array{
      arg0.bits[736]._super._super, arg0.bits[737]._super._super, arg0.bits[738]._super._super,
      arg0.bits[739]._super._super, arg0.bits[740]._super._super, arg0.bits[741]._super._super,
      arg0.bits[742]._super._super, arg0.bits[743]._super._super, arg0.bits[744]._super._super,
      arg0.bits[745]._super._super, arg0.bits[746]._super._super, arg0.bits[747]._super._super,
      arg0.bits[748]._super._super, arg0.bits[749]._super._super, arg0.bits[750]._super._super,
      arg0.bits[751]._super._super, arg0.bits[752]._super._super, arg0.bits[753]._super._super,
      arg0.bits[754]._super._super, arg0.bits[755]._super._super, arg0.bits[756]._super._super,
      arg0.bits[757]._super._super, arg0.bits[758]._super._super, arg0.bits[759]._super._super,
      arg0.bits[760]._super._super, arg0.bits[761]._super._super, arg0.bits[762]._super._super,
      arg0.bits[763]._super._super, arg0.bits[764]._super._super, arg0.bits[765]._super._super,
      arg0.bits[766]._super._super, arg0.bits[767]._super._super};
  Val32Array x14 = Val32Array{
      arg0.bits[704]._super._super, arg0.bits[705]._super._super, arg0.bits[706]._super._super,
      arg0.bits[707]._super._super, arg0.bits[708]._super._super, arg0.bits[709]._super._super,
      arg0.bits[710]._super._super, arg0.bits[711]._super._super, arg0.bits[712]._super._super,
      arg0.bits[713]._super._super, arg0.bits[714]._super._super, arg0.bits[715]._super._super,
      arg0.bits[716]._super._super, arg0.bits[717]._super._super, arg0.bits[718]._super._super,
      arg0.bits[719]._super._super, arg0.bits[720]._super._super, arg0.bits[721]._super._super,
      arg0.bits[722]._super._super, arg0.bits[723]._super._super, arg0.bits[724]._super._super,
      arg0.bits[725]._super._super, arg0.bits[726]._super._super, arg0.bits[727]._super._super,
      arg0.bits[728]._super._super, arg0.bits[729]._super._super, arg0.bits[730]._super._super,
      arg0.bits[731]._super._super, arg0.bits[732]._super._super, arg0.bits[733]._super._super,
      arg0.bits[734]._super._super, arg0.bits[735]._super._super};
  Val32Array x15 = Val32Array{
      arg0.bits[672]._super._super, arg0.bits[673]._super._super, arg0.bits[674]._super._super,
      arg0.bits[675]._super._super, arg0.bits[676]._super._super, arg0.bits[677]._super._super,
      arg0.bits[678]._super._super, arg0.bits[679]._super._super, arg0.bits[680]._super._super,
      arg0.bits[681]._super._super, arg0.bits[682]._super._super, arg0.bits[683]._super._super,
      arg0.bits[684]._super._super, arg0.bits[685]._super._super, arg0.bits[686]._super._super,
      arg0.bits[687]._super._super, arg0.bits[688]._super._super, arg0.bits[689]._super._super,
      arg0.bits[690]._super._super, arg0.bits[691]._super._super, arg0.bits[692]._super._super,
      arg0.bits[693]._super._super, arg0.bits[694]._super._super, arg0.bits[695]._super._super,
      arg0.bits[696]._super._super, arg0.bits[697]._super._super, arg0.bits[698]._super._super,
      arg0.bits[699]._super._super, arg0.bits[700]._super._super, arg0.bits[701]._super._super,
      arg0.bits[702]._super._super, arg0.bits[703]._super._super};
  Val32Array x16 = Val32Array{
      arg0.bits[640]._super._super, arg0.bits[641]._super._super, arg0.bits[642]._super._super,
      arg0.bits[643]._super._super, arg0.bits[644]._super._super, arg0.bits[645]._super._super,
      arg0.bits[646]._super._super, arg0.bits[647]._super._super, arg0.bits[648]._super._super,
      arg0.bits[649]._super._super, arg0.bits[650]._super._super, arg0.bits[651]._super._super,
      arg0.bits[652]._super._super, arg0.bits[653]._super._super, arg0.bits[654]._super._super,
      arg0.bits[655]._super._super, arg0.bits[656]._super._super, arg0.bits[657]._super._super,
      arg0.bits[658]._super._super, arg0.bits[659]._super._super, arg0.bits[660]._super._super,
      arg0.bits[661]._super._super, arg0.bits[662]._super._super, arg0.bits[663]._super._super,
      arg0.bits[664]._super._super, arg0.bits[665]._super._super, arg0.bits[666]._super._super,
      arg0.bits[667]._super._super, arg0.bits[668]._super._super, arg0.bits[669]._super._super,
      arg0.bits[670]._super._super, arg0.bits[671]._super._super};
  Val32Array x17 = Val32Array{
      arg0.bits[608]._super._super, arg0.bits[609]._super._super, arg0.bits[610]._super._super,
      arg0.bits[611]._super._super, arg0.bits[612]._super._super, arg0.bits[613]._super._super,
      arg0.bits[614]._super._super, arg0.bits[615]._super._super, arg0.bits[616]._super._super,
      arg0.bits[617]._super._super, arg0.bits[618]._super._super, arg0.bits[619]._super._super,
      arg0.bits[620]._super._super, arg0.bits[621]._super._super, arg0.bits[622]._super._super,
      arg0.bits[623]._super._super, arg0.bits[624]._super._super, arg0.bits[625]._super._super,
      arg0.bits[626]._super._super, arg0.bits[627]._super._super, arg0.bits[628]._super._super,
      arg0.bits[629]._super._super, arg0.bits[630]._super._super, arg0.bits[631]._super._super,
      arg0.bits[632]._super._super, arg0.bits[633]._super._super, arg0.bits[634]._super._super,
      arg0.bits[635]._super._super, arg0.bits[636]._super._super, arg0.bits[637]._super._super,
      arg0.bits[638]._super._super, arg0.bits[639]._super._super};
  Val32Array x18 = Val32Array{
      arg0.bits[576]._super._super, arg0.bits[577]._super._super, arg0.bits[578]._super._super,
      arg0.bits[579]._super._super, arg0.bits[580]._super._super, arg0.bits[581]._super._super,
      arg0.bits[582]._super._super, arg0.bits[583]._super._super, arg0.bits[584]._super._super,
      arg0.bits[585]._super._super, arg0.bits[586]._super._super, arg0.bits[587]._super._super,
      arg0.bits[588]._super._super, arg0.bits[589]._super._super, arg0.bits[590]._super._super,
      arg0.bits[591]._super._super, arg0.bits[592]._super._super, arg0.bits[593]._super._super,
      arg0.bits[594]._super._super, arg0.bits[595]._super._super, arg0.bits[596]._super._super,
      arg0.bits[597]._super._super, arg0.bits[598]._super._super, arg0.bits[599]._super._super,
      arg0.bits[600]._super._super, arg0.bits[601]._super._super, arg0.bits[602]._super._super,
      arg0.bits[603]._super._super, arg0.bits[604]._super._super, arg0.bits[605]._super._super,
      arg0.bits[606]._super._super, arg0.bits[607]._super._super};
  Val32Array x19 = Val32Array{
      arg0.bits[544]._super._super, arg0.bits[545]._super._super, arg0.bits[546]._super._super,
      arg0.bits[547]._super._super, arg0.bits[548]._super._super, arg0.bits[549]._super._super,
      arg0.bits[550]._super._super, arg0.bits[551]._super._super, arg0.bits[552]._super._super,
      arg0.bits[553]._super._super, arg0.bits[554]._super._super, arg0.bits[555]._super._super,
      arg0.bits[556]._super._super, arg0.bits[557]._super._super, arg0.bits[558]._super._super,
      arg0.bits[559]._super._super, arg0.bits[560]._super._super, arg0.bits[561]._super._super,
      arg0.bits[562]._super._super, arg0.bits[563]._super._super, arg0.bits[564]._super._super,
      arg0.bits[565]._super._super, arg0.bits[566]._super._super, arg0.bits[567]._super._super,
      arg0.bits[568]._super._super, arg0.bits[569]._super._super, arg0.bits[570]._super._super,
      arg0.bits[571]._super._super, arg0.bits[572]._super._super, arg0.bits[573]._super._super,
      arg0.bits[574]._super._super, arg0.bits[575]._super._super};
  Val32Array x20 = Val32Array{
      arg0.bits[512]._super._super, arg0.bits[513]._super._super, arg0.bits[514]._super._super,
      arg0.bits[515]._super._super, arg0.bits[516]._super._super, arg0.bits[517]._super._super,
      arg0.bits[518]._super._super, arg0.bits[519]._super._super, arg0.bits[520]._super._super,
      arg0.bits[521]._super._super, arg0.bits[522]._super._super, arg0.bits[523]._super._super,
      arg0.bits[524]._super._super, arg0.bits[525]._super._super, arg0.bits[526]._super._super,
      arg0.bits[527]._super._super, arg0.bits[528]._super._super, arg0.bits[529]._super._super,
      arg0.bits[530]._super._super, arg0.bits[531]._super._super, arg0.bits[532]._super._super,
      arg0.bits[533]._super._super, arg0.bits[534]._super._super, arg0.bits[535]._super._super,
      arg0.bits[536]._super._super, arg0.bits[537]._super._super, arg0.bits[538]._super._super,
      arg0.bits[539]._super._super, arg0.bits[540]._super._super, arg0.bits[541]._super._super,
      arg0.bits[542]._super._super, arg0.bits[543]._super._super};
  Val32Array x21 = Val32Array{
      arg1.bits[736]._super._super, arg1.bits[737]._super._super, arg1.bits[738]._super._super,
      arg1.bits[739]._super._super, arg1.bits[740]._super._super, arg1.bits[741]._super._super,
      arg1.bits[742]._super._super, arg1.bits[743]._super._super, arg1.bits[744]._super._super,
      arg1.bits[745]._super._super, arg1.bits[746]._super._super, arg1.bits[747]._super._super,
      arg1.bits[748]._super._super, arg1.bits[749]._super._super, arg1.bits[750]._super._super,
      arg1.bits[751]._super._super, arg1.bits[752]._super._super, arg1.bits[753]._super._super,
      arg1.bits[754]._super._super, arg1.bits[755]._super._super, arg1.bits[756]._super._super,
      arg1.bits[757]._super._super, arg1.bits[758]._super._super, arg1.bits[759]._super._super,
      arg1.bits[760]._super._super, arg1.bits[761]._super._super, arg1.bits[762]._super._super,
      arg1.bits[763]._super._super, arg1.bits[764]._super._super, arg1.bits[765]._super._super,
      arg1.bits[766]._super._super, arg1.bits[767]._super._super};
  Val32Array x22 = Val32Array{
      arg1.bits[704]._super._super, arg1.bits[705]._super._super, arg1.bits[706]._super._super,
      arg1.bits[707]._super._super, arg1.bits[708]._super._super, arg1.bits[709]._super._super,
      arg1.bits[710]._super._super, arg1.bits[711]._super._super, arg1.bits[712]._super._super,
      arg1.bits[713]._super._super, arg1.bits[714]._super._super, arg1.bits[715]._super._super,
      arg1.bits[716]._super._super, arg1.bits[717]._super._super, arg1.bits[718]._super._super,
      arg1.bits[719]._super._super, arg1.bits[720]._super._super, arg1.bits[721]._super._super,
      arg1.bits[722]._super._super, arg1.bits[723]._super._super, arg1.bits[724]._super._super,
      arg1.bits[725]._super._super, arg1.bits[726]._super._super, arg1.bits[727]._super._super,
      arg1.bits[728]._super._super, arg1.bits[729]._super._super, arg1.bits[730]._super._super,
      arg1.bits[731]._super._super, arg1.bits[732]._super._super, arg1.bits[733]._super._super,
      arg1.bits[734]._super._super, arg1.bits[735]._super._super};
  Val32Array x23 = Val32Array{
      arg1.bits[672]._super._super, arg1.bits[673]._super._super, arg1.bits[674]._super._super,
      arg1.bits[675]._super._super, arg1.bits[676]._super._super, arg1.bits[677]._super._super,
      arg1.bits[678]._super._super, arg1.bits[679]._super._super, arg1.bits[680]._super._super,
      arg1.bits[681]._super._super, arg1.bits[682]._super._super, arg1.bits[683]._super._super,
      arg1.bits[684]._super._super, arg1.bits[685]._super._super, arg1.bits[686]._super._super,
      arg1.bits[687]._super._super, arg1.bits[688]._super._super, arg1.bits[689]._super._super,
      arg1.bits[690]._super._super, arg1.bits[691]._super._super, arg1.bits[692]._super._super,
      arg1.bits[693]._super._super, arg1.bits[694]._super._super, arg1.bits[695]._super._super,
      arg1.bits[696]._super._super, arg1.bits[697]._super._super, arg1.bits[698]._super._super,
      arg1.bits[699]._super._super, arg1.bits[700]._super._super, arg1.bits[701]._super._super,
      arg1.bits[702]._super._super, arg1.bits[703]._super._super};
  Val32Array x24 = Val32Array{
      arg1.bits[640]._super._super, arg1.bits[641]._super._super, arg1.bits[642]._super._super,
      arg1.bits[643]._super._super, arg1.bits[644]._super._super, arg1.bits[645]._super._super,
      arg1.bits[646]._super._super, arg1.bits[647]._super._super, arg1.bits[648]._super._super,
      arg1.bits[649]._super._super, arg1.bits[650]._super._super, arg1.bits[651]._super._super,
      arg1.bits[652]._super._super, arg1.bits[653]._super._super, arg1.bits[654]._super._super,
      arg1.bits[655]._super._super, arg1.bits[656]._super._super, arg1.bits[657]._super._super,
      arg1.bits[658]._super._super, arg1.bits[659]._super._super, arg1.bits[660]._super._super,
      arg1.bits[661]._super._super, arg1.bits[662]._super._super, arg1.bits[663]._super._super,
      arg1.bits[664]._super._super, arg1.bits[665]._super._super, arg1.bits[666]._super._super,
      arg1.bits[667]._super._super, arg1.bits[668]._super._super, arg1.bits[669]._super._super,
      arg1.bits[670]._super._super, arg1.bits[671]._super._super};
  Val32Array x25 = Val32Array{
      arg1.bits[608]._super._super, arg1.bits[609]._super._super, arg1.bits[610]._super._super,
      arg1.bits[611]._super._super, arg1.bits[612]._super._super, arg1.bits[613]._super._super,
      arg1.bits[614]._super._super, arg1.bits[615]._super._super, arg1.bits[616]._super._super,
      arg1.bits[617]._super._super, arg1.bits[618]._super._super, arg1.bits[619]._super._super,
      arg1.bits[620]._super._super, arg1.bits[621]._super._super, arg1.bits[622]._super._super,
      arg1.bits[623]._super._super, arg1.bits[624]._super._super, arg1.bits[625]._super._super,
      arg1.bits[626]._super._super, arg1.bits[627]._super._super, arg1.bits[628]._super._super,
      arg1.bits[629]._super._super, arg1.bits[630]._super._super, arg1.bits[631]._super._super,
      arg1.bits[632]._super._super, arg1.bits[633]._super._super, arg1.bits[634]._super._super,
      arg1.bits[635]._super._super, arg1.bits[636]._super._super, arg1.bits[637]._super._super,
      arg1.bits[638]._super._super, arg1.bits[639]._super._super};
  Val32Array x26 = Val32Array{
      arg1.bits[576]._super._super, arg1.bits[577]._super._super, arg1.bits[578]._super._super,
      arg1.bits[579]._super._super, arg1.bits[580]._super._super, arg1.bits[581]._super._super,
      arg1.bits[582]._super._super, arg1.bits[583]._super._super, arg1.bits[584]._super._super,
      arg1.bits[585]._super._super, arg1.bits[586]._super._super, arg1.bits[587]._super._super,
      arg1.bits[588]._super._super, arg1.bits[589]._super._super, arg1.bits[590]._super._super,
      arg1.bits[591]._super._super, arg1.bits[592]._super._super, arg1.bits[593]._super._super,
      arg1.bits[594]._super._super, arg1.bits[595]._super._super, arg1.bits[596]._super._super,
      arg1.bits[597]._super._super, arg1.bits[598]._super._super, arg1.bits[599]._super._super,
      arg1.bits[600]._super._super, arg1.bits[601]._super._super, arg1.bits[602]._super._super,
      arg1.bits[603]._super._super, arg1.bits[604]._super._super, arg1.bits[605]._super._super,
      arg1.bits[606]._super._super, arg1.bits[607]._super._super};
  Val32Array x27 = Val32Array{
      arg1.bits[544]._super._super, arg1.bits[545]._super._super, arg1.bits[546]._super._super,
      arg1.bits[547]._super._super, arg1.bits[548]._super._super, arg1.bits[549]._super._super,
      arg1.bits[550]._super._super, arg1.bits[551]._super._super, arg1.bits[552]._super._super,
      arg1.bits[553]._super._super, arg1.bits[554]._super._super, arg1.bits[555]._super._super,
      arg1.bits[556]._super._super, arg1.bits[557]._super._super, arg1.bits[558]._super._super,
      arg1.bits[559]._super._super, arg1.bits[560]._super._super, arg1.bits[561]._super._super,
      arg1.bits[562]._super._super, arg1.bits[563]._super._super, arg1.bits[564]._super._super,
      arg1.bits[565]._super._super, arg1.bits[566]._super._super, arg1.bits[567]._super._super,
      arg1.bits[568]._super._super, arg1.bits[569]._super._super, arg1.bits[570]._super._super,
      arg1.bits[571]._super._super, arg1.bits[572]._super._super, arg1.bits[573]._super._super,
      arg1.bits[574]._super._super, arg1.bits[575]._super._super};
  Val32Array x28 = Val32Array{
      arg1.bits[512]._super._super, arg1.bits[513]._super._super, arg1.bits[514]._super._super,
      arg1.bits[515]._super._super, arg1.bits[516]._super._super, arg1.bits[517]._super._super,
      arg1.bits[518]._super._super, arg1.bits[519]._super._super, arg1.bits[520]._super._super,
      arg1.bits[521]._super._super, arg1.bits[522]._super._super, arg1.bits[523]._super._super,
      arg1.bits[524]._super._super, arg1.bits[525]._super._super, arg1.bits[526]._super._super,
      arg1.bits[527]._super._super, arg1.bits[528]._super._super, arg1.bits[529]._super._super,
      arg1.bits[530]._super._super, arg1.bits[531]._super._super, arg1.bits[532]._super._super,
      arg1.bits[533]._super._super, arg1.bits[534]._super._super, arg1.bits[535]._super._super,
      arg1.bits[536]._super._super, arg1.bits[537]._super._super, arg1.bits[538]._super._super,
      arg1.bits[539]._super._super, arg1.bits[540]._super._super, arg1.bits[541]._super._super,
      arg1.bits[542]._super._super, arg1.bits[543]._super._super};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:190)
  OneHot_8_Struct x29 = exec_OneHot_8_(ctx, arg2, LAYOUT_LOOKUP(layout4, oneHot));
  Val x30;
  if (to_size_t(x29._super[0]._super)) {
    x30 = Val(1);
  } else if (to_size_t(x29._super[1]._super)) {
    x30 = Val(1);
  } else if (to_size_t(x29._super[2]._super)) {
    x30 = Val(0);
  } else if (to_size_t(x29._super[3]._super)) {
    x30 = Val(0);
  } else if (to_size_t(x29._super[4]._super)) {
    x30 = Val(0);
  } else if (to_size_t(x29._super[5]._super)) {
    x30 = Val(0);
  } else if (to_size_t(x29._super[6]._super)) {
    x30 = Val(0);
  } else if (to_size_t(x29._super[7]._super)) {
    x30 = Val(0);
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  // ShaCycle(zirgen/circuit/keccak2/top.zir:191)
  NondetRegStruct x31 = exec_Reg(ctx, x30, LAYOUT_LOOKUP(layout4, isLoad));
  // GetK8(zirgen/circuit/keccak2/top.zir:125)
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val x32 = (x29._super[0]._super * Val(12184));
  Val x33 = (x29._super[1]._super * Val(43672));
  Val x34 = (x29._super[2]._super * Val(27073));
  Val x35 = (x29._super[3]._super * Val(20818));
  Val x36 = (x29._super[4]._super * Val(2693));
  Val x37 = (x29._super[5]._super * Val(59553));
  Val x38 = (x29._super[6]._super * Val(49430));
  Val x39 = (x29._super[7]._super * Val(33518));
  Val x40 = (((x32 + x33) + x34) + x35);
  Val x41 = (((x40 + x36) + x37) + x38);
  Val x42 = (x29._super[0]._super * Val(17034));
  Val x43 = (x29._super[1]._super * Val(55303));
  Val x44 = (x29._super[2]._super * Val(58523));
  Val x45 = (x29._super[3]._super * Val(38974));
  Val x46 = (x29._super[4]._super * Val(10167));
  Val x47 = (x29._super[5]._super * Val(41663));
  Val x48 = (x29._super[6]._super * Val(6564));
  Val x49 = (x29._super[7]._super * Val(29839));
  Val x50 = (((x42 + x43) + x44) + x45);
  Val x51 = (((x50 + x46) + x47) + x48);
  Val x52 = (x29._super[0]._super * Val(17553));
  Val x53 = (x29._super[1]._super * Val(23297));
  Val x54 = (x29._super[2]._super * Val(18310));
  Val x55 = (x29._super[3]._super * Val(50797));
  Val x56 = (x29._super[4]._super * Val(8504));
  Val x57 = (x29._super[5]._super * Val(26187));
  Val x58 = (x29._super[6]._super * Val(27656));
  Val x59 = (x29._super[7]._super * Val(25455));
  Val x60 = (((x52 + x53) + x54) + x55);
  Val x61 = (((x60 + x56) + x57) + x58);
  Val x62 = (x29._super[0]._super * Val(28983));
  Val x63 = (x29._super[1]._super * Val(4739));
  Val x64 = (x29._super[2]._super * Val(61374));
  Val x65 = (x29._super[3]._super * Val(43057));
  Val x66 = (x29._super[4]._super * Val(11803));
  Val x67 = (x29._super[5]._super * Val(43034));
  Val x68 = (x29._super[6]._super * Val(7735));
  Val x69 = (x29._super[7]._super * Val(30885));
  Val x70 = (((x62 + x63) + x64) + x65);
  Val x71 = (((x70 + x66) + x67) + x68);
  Val x72 = (x29._super[0]._super * Val(64463));
  Val x73 = (x29._super[1]._super * Val(34238));
  Val x74 = (x29._super[2]._super * Val(40390));
  Val x75 = (x29._super[3]._super * Val(10184));
  Val x76 = (x29._super[4]._super * Val(28156));
  Val x77 = (x29._super[5]._super * Val(35696));
  Val x78 = (x29._super[6]._super * Val(30540));
  Val x79 = (x29._super[7]._super * Val(30740));
  Val x80 = (((x72 + x73) + x74) + x75);
  Val x81 = (((x80 + x76) + x77) + x78);
  Val x82 = (x29._super[0]._super * Val(46528));
  Val x83 = (x29._super[1]._super * Val(9265));
  Val x84 = (x29._super[2]._super * Val(4033));
  Val x85 = (x29._super[3]._super * Val(45059));
  Val x86 = (x29._super[4]._super * Val(19756));
  Val x87 = (x29._super[5]._super * Val(49739));
  Val x88 = (x29._super[6]._super * Val(10056));
  Val x89 = (x29._super[7]._super * Val(33992));
  Val x90 = (((x82 + x83) + x84) + x85);
  Val x91 = (((x90 + x86) + x87) + x88);
  Val x92 = (x29._super[0]._super * Val(56229));
  Val x93 = (x29._super[1]._super * Val(32195));
  Val x94 = (x29._super[2]._super * Val(41420));
  Val x95 = (x29._super[3]._super * Val(32711));
  Val x96 = (x29._super[4]._super * Val(3347));
  Val x97 = (x29._super[5]._super * Val(20899));
  Val x98 = (x29._super[6]._super * Val(48309));
  Val x99 = (x29._super[7]._super * Val(520));
  Val x100 = (((x92 + x93) + x94) + x95);
  Val x101 = (((x100 + x96) + x97) + x98);
  Val x102 = (x29._super[0]._super * Val(59829));
  Val x103 = (x29._super[1]._super * Val(21772));
  Val x104 = (x29._super[2]._super * Val(9228));
  Val x105 = (x29._super[3]._super * Val(48985));
  Val x106 = (x29._super[4]._super * Val(21304));
  Val x107 = (x29._super[5]._super * Val(51052));
  Val x108 = (x29._super[6]._super * Val(13488));
  Val x109 = (x29._super[7]._super * Val(36039));
  Val x110 = (((x102 + x103) + x104) + x105);
  Val x111 = (((x110 + x106) + x107) + x108);
  Val x112 = (x29._super[0]._super * Val(49755));
  Val x113 = (x29._super[1]._super * Val(23924));
  Val x114 = (x29._super[2]._super * Val(11375));
  Val x115 = (x29._super[3]._super * Val(3059));
  Val x116 = (x29._super[4]._super * Val(29524));
  Val x117 = (x29._super[5]._super * Val(59417));
  Val x118 = (x29._super[6]._super * Val(3251));
  Val x119 = (x29._super[7]._super * Val(65530));
  Val x120 = (((x112 + x113) + x114) + x115);
  Val x121 = (((x120 + x116) + x117) + x118);
  Val x122 = (x29._super[0]._super * Val(14678));
  Val x123 = (x29._super[1]._super * Val(29374));
  Val x124 = (x29._super[2]._super * Val(11753));
  Val x125 = (x29._super[3]._super * Val(50912));
  Val x126 = (x29._super[4]._super * Val(25866));
  Val x127 = (x29._super[5]._super * Val(53650));
  Val x128 = (x29._super[6]._super * Val(14620));
  Val x129 = (x29._super[7]._super * Val(37054));
  Val x130 = (((x122 + x123) + x124) + x125);
  Val x131 = (((x130 + x126) + x127) + x128);
  Val x132 = (x29._super[0]._super * Val(4593));
  Val x133 = (x29._super[1]._super * Val(45566));
  Val x134 = (x29._super[2]._super * Val(33962));
  Val x135 = (x29._super[3]._super * Val(37191));
  Val x136 = (x29._super[4]._super * Val(2747));
  Val x137 = (x29._super[5]._super * Val(1572));
  Val x138 = (x29._super[6]._super * Val(43594));
  Val x139 = (x29._super[7]._super * Val(27883));
  Val x140 = (((x132 + x133) + x134) + x135);
  Val x141 = (((x140 + x136) + x137) + x138);
  Val x142 = (x29._super[0]._super * Val(23025));
  Val x143 = (x29._super[1]._super * Val(32990));
  Val x144 = (x29._super[2]._super * Val(19060));
  Val x145 = (x29._super[3]._super * Val(54695));
  Val x146 = (x29._super[4]._super * Val(30314));
  Val x147 = (x29._super[5]._super * Val(54937));
  Val x148 = (x29._super[6]._super * Val(20184));
  Val x149 = (x29._super[7]._super * Val(42064));
  Val x150 = (((x142 + x143) + x144) + x145);
  Val x151 = (((x150 + x146) + x147) + x148);
  Val x152 = (x29._super[0]._super * Val(33444));
  Val x153 = (x29._super[1]._super * Val(1703));
  Val x154 = (x29._super[2]._super * Val(43484));
  Val x155 = (x29._super[3]._super * Val(25425));
  Val x156 = (x29._super[4]._super * Val(51502));
  Val x157 = (x29._super[5]._super * Val(13701));
  Val x158 = (x29._super[6]._super * Val(51791));
  Val x159 = (x29._super[7]._super * Val(41975));
  Val x160 = (((x152 + x153) + x154) + x155);
  Val x161 = (((x160 + x156) + x157) + x158);
  Val x162 = (x29._super[0]._super * Val(37439));
  Val x163 = (x29._super[1]._super * Val(39900));
  Val x164 = (x29._super[2]._super * Val(23728));
  Val x165 = (x29._super[3]._super * Val(1738));
  Val x166 = (x29._super[4]._super * Val(33218));
  Val x167 = (x29._super[5]._super * Val(62478));
  Val x168 = (x29._super[6]._super * Val(23452));
  Val x169 = (x29._super[7]._super * Val(48889));
  Val x170 = (((x162 + x163) + x164) + x165);
  Val x171 = (((x170 + x166) + x167) + x168);
  Val x172 = (x29._super[0]._super * Val(24277));
  Val x173 = (x29._super[1]._super * Val(61812));
  Val x174 = (x29._super[2]._super * Val(35034));
  Val x175 = (x29._super[3]._super * Val(10599));
  Val x176 = (x29._super[4]._super * Val(11397));
  Val x177 = (x29._super[5]._super * Val(41072));
  Val x178 = (x29._super[6]._super * Val(28659));
  Val x179 = (x29._super[7]._super * Val(30962));
  Val x180 = (((x172 + x173) + x174) + x175);
  Val x181 = (((x180 + x176) + x177) + x178);
  Val x182 = (x29._super[0]._super * Val(43804));
  Val x183 = (x29._super[1]._super * Val(49563));
  Val x184 = (x29._super[2]._super * Val(30457));
  Val x185 = (x29._super[3]._super * Val(5161));
  Val x186 = (x29._super[4]._super * Val(37490));
  Val x187 = (x29._super[5]._super * Val(4202));
  Val x188 = (x29._super[6]._super * Val(26670));
  Val x189 = (x29._super[7]._super * Val(50801));
  Val x190 = (((x182 + x183) + x184) + x185);
  Val x191 = (((x190 + x186) + x187) + x188);
  // ShaCycle(zirgen/circuit/keccak2/top.zir:186)
  Val100Array x192 = Val100Array{
      arg0.kflat[0]._super._super,  arg0.kflat[1]._super._super,  arg0.kflat[2]._super._super,
      arg0.kflat[3]._super._super,  arg0.kflat[4]._super._super,  arg0.kflat[5]._super._super,
      arg0.kflat[6]._super._super,  arg0.kflat[7]._super._super,  arg0.kflat[8]._super._super,
      arg0.kflat[9]._super._super,  arg0.kflat[10]._super._super, arg0.kflat[11]._super._super,
      arg0.kflat[12]._super._super, arg0.kflat[13]._super._super, arg0.kflat[14]._super._super,
      arg0.kflat[15]._super._super, arg0.kflat[16]._super._super, arg0.kflat[17]._super._super,
      arg0.kflat[18]._super._super, arg0.kflat[19]._super._super, arg0.kflat[20]._super._super,
      arg0.kflat[21]._super._super, arg0.kflat[22]._super._super, arg0.kflat[23]._super._super,
      arg0.kflat[24]._super._super, arg0.kflat[25]._super._super, arg0.kflat[26]._super._super,
      arg0.kflat[27]._super._super, arg0.kflat[28]._super._super, arg0.kflat[29]._super._super,
      arg0.kflat[30]._super._super, arg0.kflat[31]._super._super, arg0.kflat[32]._super._super,
      arg0.kflat[33]._super._super, arg0.kflat[34]._super._super, arg0.kflat[35]._super._super,
      arg0.kflat[36]._super._super, arg0.kflat[37]._super._super, arg0.kflat[38]._super._super,
      arg0.kflat[39]._super._super, arg0.kflat[40]._super._super, arg0.kflat[41]._super._super,
      arg0.kflat[42]._super._super, arg0.kflat[43]._super._super, arg0.kflat[44]._super._super,
      arg0.kflat[45]._super._super, arg0.kflat[46]._super._super, arg0.kflat[47]._super._super,
      arg0.kflat[48]._super._super, arg0.kflat[49]._super._super, arg0.kflat[50]._super._super,
      arg0.kflat[51]._super._super, arg0.kflat[52]._super._super, arg0.kflat[53]._super._super,
      arg0.kflat[54]._super._super, arg0.kflat[55]._super._super, arg0.kflat[56]._super._super,
      arg0.kflat[57]._super._super, arg0.kflat[58]._super._super, arg0.kflat[59]._super._super,
      arg0.kflat[60]._super._super, arg0.kflat[61]._super._super, arg0.kflat[62]._super._super,
      arg0.kflat[63]._super._super, arg0.kflat[64]._super._super, arg0.kflat[65]._super._super,
      arg0.kflat[66]._super._super, arg0.kflat[67]._super._super, arg0.kflat[68]._super._super,
      arg0.kflat[69]._super._super, arg0.kflat[70]._super._super, arg0.kflat[71]._super._super,
      arg0.kflat[72]._super._super, arg0.kflat[73]._super._super, arg0.kflat[74]._super._super,
      arg0.kflat[75]._super._super, arg0.kflat[76]._super._super, arg0.kflat[77]._super._super,
      arg0.kflat[78]._super._super, arg0.kflat[79]._super._super, arg0.kflat[80]._super._super,
      arg0.kflat[81]._super._super, arg0.kflat[82]._super._super, arg0.kflat[83]._super._super,
      arg0.kflat[84]._super._super, arg0.kflat[85]._super._super, arg0.kflat[86]._super._super,
      arg0.kflat[87]._super._super, arg0.kflat[88]._super._super, arg0.kflat[89]._super._super,
      arg0.kflat[90]._super._super, arg0.kflat[91]._super._super, arg0.kflat[92]._super._super,
      arg0.kflat[93]._super._super, arg0.kflat[94]._super._super, arg0.kflat[95]._super._super,
      arg0.kflat[96]._super._super, arg0.kflat[97]._super._super, arg0.kflat[98]._super._super,
      arg0.kflat[99]._super._super};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:193)
  LoadWinStruct x193 = exec_LoadWin(ctx, x192, x29, arg3, LAYOUT_LOOKUP(layout4, win));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:194)
  DoShaStepStruct x194 = exec_DoShaStep(
      ctx,
      ShaStateStruct{
          .a = Val32Array4Array{x5, x6, x7, x8},
          .e = Val32Array4Array{x9, x10, x11, x12},
          .w =
              Val32Array16Array{
                  x13, x14, x15, x16, x17, x18, x19, x20, x21, x22, x23, x24, x25, x26, x27, x28}},
      Val2Array{(x41 + x39), (x51 + x49)},
      x31._super,
      x193._super[0],
      LAYOUT_LOOKUP(layout4, step0));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:195)
  DoShaStepStruct x195 = exec_DoShaStep(ctx,
                                        x194.newState,
                                        Val2Array{(x61 + x59), (x71 + x69)},
                                        x31._super,
                                        x193._super[1],
                                        LAYOUT_LOOKUP(layout4, step1));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:196)
  DoShaStepStruct x196 = exec_DoShaStep(ctx,
                                        x195.newState,
                                        Val2Array{(x81 + x79), (x91 + x89)},
                                        x31._super,
                                        x193._super[2],
                                        LAYOUT_LOOKUP(layout4, step2));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:197)
  DoShaStepStruct x197 = exec_DoShaStep(ctx,
                                        x196.newState,
                                        Val2Array{(x101 + x99), (x111 + x109)},
                                        x31._super,
                                        x193._super[3],
                                        LAYOUT_LOOKUP(layout4, step3));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:198)
  DoShaStepStruct x198 = exec_DoShaStep(ctx,
                                        x197.newState,
                                        Val2Array{(x121 + x119), (x131 + x129)},
                                        x31._super,
                                        x193._super[4],
                                        LAYOUT_LOOKUP(layout4, step4));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:199)
  DoShaStepStruct x199 = exec_DoShaStep(ctx,
                                        x198.newState,
                                        Val2Array{(x141 + x139), (x151 + x149)},
                                        x31._super,
                                        x193._super[5],
                                        LAYOUT_LOOKUP(layout4, step5));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:200)
  DoShaStepStruct x200 = exec_DoShaStep(ctx,
                                        x199.newState,
                                        Val2Array{(x161 + x159), (x171 + x169)},
                                        x31._super,
                                        x193._super[6],
                                        LAYOUT_LOOKUP(layout4, step6));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:201)
  DoShaStepStruct x201 = exec_DoShaStep(ctx,
                                        x200.newState,
                                        Val2Array{(x181 + x179), (x191 + x189)},
                                        x31._super,
                                        x193._super[7],
                                        LAYOUT_LOOKUP(layout4, step7));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:203)
  Val800Array x202 = Val800Array{x194.a._super[0]._super._super,
                                 x194.a._super[1]._super._super,
                                 x194.a._super[2]._super._super,
                                 x194.a._super[3]._super._super,
                                 x194.a._super[4]._super._super,
                                 x194.a._super[5]._super._super,
                                 x194.a._super[6]._super._super,
                                 x194.a._super[7]._super._super,
                                 x194.a._super[8]._super._super,
                                 x194.a._super[9]._super._super,
                                 x194.a._super[10]._super._super,
                                 x194.a._super[11]._super._super,
                                 x194.a._super[12]._super._super,
                                 x194.a._super[13]._super._super,
                                 x194.a._super[14]._super._super,
                                 x194.a._super[15]._super._super,
                                 x194.a._super[16]._super._super,
                                 x194.a._super[17]._super._super,
                                 x194.a._super[18]._super._super,
                                 x194.a._super[19]._super._super,
                                 x194.a._super[20]._super._super,
                                 x194.a._super[21]._super._super,
                                 x194.a._super[22]._super._super,
                                 x194.a._super[23]._super._super,
                                 x194.a._super[24]._super._super,
                                 x194.a._super[25]._super._super,
                                 x194.a._super[26]._super._super,
                                 x194.a._super[27]._super._super,
                                 x194.a._super[28]._super._super,
                                 x194.a._super[29]._super._super,
                                 x194.a._super[30]._super._super,
                                 x194.a._super[31]._super._super,
                                 x195.a._super[0]._super._super,
                                 x195.a._super[1]._super._super,
                                 x195.a._super[2]._super._super,
                                 x195.a._super[3]._super._super,
                                 x195.a._super[4]._super._super,
                                 x195.a._super[5]._super._super,
                                 x195.a._super[6]._super._super,
                                 x195.a._super[7]._super._super,
                                 x195.a._super[8]._super._super,
                                 x195.a._super[9]._super._super,
                                 x195.a._super[10]._super._super,
                                 x195.a._super[11]._super._super,
                                 x195.a._super[12]._super._super,
                                 x195.a._super[13]._super._super,
                                 x195.a._super[14]._super._super,
                                 x195.a._super[15]._super._super,
                                 x195.a._super[16]._super._super,
                                 x195.a._super[17]._super._super,
                                 x195.a._super[18]._super._super,
                                 x195.a._super[19]._super._super,
                                 x195.a._super[20]._super._super,
                                 x195.a._super[21]._super._super,
                                 x195.a._super[22]._super._super,
                                 x195.a._super[23]._super._super,
                                 x195.a._super[24]._super._super,
                                 x195.a._super[25]._super._super,
                                 x195.a._super[26]._super._super,
                                 x195.a._super[27]._super._super,
                                 x195.a._super[28]._super._super,
                                 x195.a._super[29]._super._super,
                                 x195.a._super[30]._super._super,
                                 x195.a._super[31]._super._super,
                                 x196.a._super[0]._super._super,
                                 x196.a._super[1]._super._super,
                                 x196.a._super[2]._super._super,
                                 x196.a._super[3]._super._super,
                                 x196.a._super[4]._super._super,
                                 x196.a._super[5]._super._super,
                                 x196.a._super[6]._super._super,
                                 x196.a._super[7]._super._super,
                                 x196.a._super[8]._super._super,
                                 x196.a._super[9]._super._super,
                                 x196.a._super[10]._super._super,
                                 x196.a._super[11]._super._super,
                                 x196.a._super[12]._super._super,
                                 x196.a._super[13]._super._super,
                                 x196.a._super[14]._super._super,
                                 x196.a._super[15]._super._super,
                                 x196.a._super[16]._super._super,
                                 x196.a._super[17]._super._super,
                                 x196.a._super[18]._super._super,
                                 x196.a._super[19]._super._super,
                                 x196.a._super[20]._super._super,
                                 x196.a._super[21]._super._super,
                                 x196.a._super[22]._super._super,
                                 x196.a._super[23]._super._super,
                                 x196.a._super[24]._super._super,
                                 x196.a._super[25]._super._super,
                                 x196.a._super[26]._super._super,
                                 x196.a._super[27]._super._super,
                                 x196.a._super[28]._super._super,
                                 x196.a._super[29]._super._super,
                                 x196.a._super[30]._super._super,
                                 x196.a._super[31]._super._super,
                                 x197.a._super[0]._super._super,
                                 x197.a._super[1]._super._super,
                                 x197.a._super[2]._super._super,
                                 x197.a._super[3]._super._super,
                                 x197.a._super[4]._super._super,
                                 x197.a._super[5]._super._super,
                                 x197.a._super[6]._super._super,
                                 x197.a._super[7]._super._super,
                                 x197.a._super[8]._super._super,
                                 x197.a._super[9]._super._super,
                                 x197.a._super[10]._super._super,
                                 x197.a._super[11]._super._super,
                                 x197.a._super[12]._super._super,
                                 x197.a._super[13]._super._super,
                                 x197.a._super[14]._super._super,
                                 x197.a._super[15]._super._super,
                                 x197.a._super[16]._super._super,
                                 x197.a._super[17]._super._super,
                                 x197.a._super[18]._super._super,
                                 x197.a._super[19]._super._super,
                                 x197.a._super[20]._super._super,
                                 x197.a._super[21]._super._super,
                                 x197.a._super[22]._super._super,
                                 x197.a._super[23]._super._super,
                                 x197.a._super[24]._super._super,
                                 x197.a._super[25]._super._super,
                                 x197.a._super[26]._super._super,
                                 x197.a._super[27]._super._super,
                                 x197.a._super[28]._super._super,
                                 x197.a._super[29]._super._super,
                                 x197.a._super[30]._super._super,
                                 x197.a._super[31]._super._super,
                                 x198.a._super[0]._super._super,
                                 x198.a._super[1]._super._super,
                                 x198.a._super[2]._super._super,
                                 x198.a._super[3]._super._super,
                                 x198.a._super[4]._super._super,
                                 x198.a._super[5]._super._super,
                                 x198.a._super[6]._super._super,
                                 x198.a._super[7]._super._super,
                                 x198.a._super[8]._super._super,
                                 x198.a._super[9]._super._super,
                                 x198.a._super[10]._super._super,
                                 x198.a._super[11]._super._super,
                                 x198.a._super[12]._super._super,
                                 x198.a._super[13]._super._super,
                                 x198.a._super[14]._super._super,
                                 x198.a._super[15]._super._super,
                                 x198.a._super[16]._super._super,
                                 x198.a._super[17]._super._super,
                                 x198.a._super[18]._super._super,
                                 x198.a._super[19]._super._super,
                                 x198.a._super[20]._super._super,
                                 x198.a._super[21]._super._super,
                                 x198.a._super[22]._super._super,
                                 x198.a._super[23]._super._super,
                                 x198.a._super[24]._super._super,
                                 x198.a._super[25]._super._super,
                                 x198.a._super[26]._super._super,
                                 x198.a._super[27]._super._super,
                                 x198.a._super[28]._super._super,
                                 x198.a._super[29]._super._super,
                                 x198.a._super[30]._super._super,
                                 x198.a._super[31]._super._super,
                                 x199.a._super[0]._super._super,
                                 x199.a._super[1]._super._super,
                                 x199.a._super[2]._super._super,
                                 x199.a._super[3]._super._super,
                                 x199.a._super[4]._super._super,
                                 x199.a._super[5]._super._super,
                                 x199.a._super[6]._super._super,
                                 x199.a._super[7]._super._super,
                                 x199.a._super[8]._super._super,
                                 x199.a._super[9]._super._super,
                                 x199.a._super[10]._super._super,
                                 x199.a._super[11]._super._super,
                                 x199.a._super[12]._super._super,
                                 x199.a._super[13]._super._super,
                                 x199.a._super[14]._super._super,
                                 x199.a._super[15]._super._super,
                                 x199.a._super[16]._super._super,
                                 x199.a._super[17]._super._super,
                                 x199.a._super[18]._super._super,
                                 x199.a._super[19]._super._super,
                                 x199.a._super[20]._super._super,
                                 x199.a._super[21]._super._super,
                                 x199.a._super[22]._super._super,
                                 x199.a._super[23]._super._super,
                                 x199.a._super[24]._super._super,
                                 x199.a._super[25]._super._super,
                                 x199.a._super[26]._super._super,
                                 x199.a._super[27]._super._super,
                                 x199.a._super[28]._super._super,
                                 x199.a._super[29]._super._super,
                                 x199.a._super[30]._super._super,
                                 x199.a._super[31]._super._super,
                                 x200.a._super[0]._super._super,
                                 x200.a._super[1]._super._super,
                                 x200.a._super[2]._super._super,
                                 x200.a._super[3]._super._super,
                                 x200.a._super[4]._super._super,
                                 x200.a._super[5]._super._super,
                                 x200.a._super[6]._super._super,
                                 x200.a._super[7]._super._super,
                                 x200.a._super[8]._super._super,
                                 x200.a._super[9]._super._super,
                                 x200.a._super[10]._super._super,
                                 x200.a._super[11]._super._super,
                                 x200.a._super[12]._super._super,
                                 x200.a._super[13]._super._super,
                                 x200.a._super[14]._super._super,
                                 x200.a._super[15]._super._super,
                                 x200.a._super[16]._super._super,
                                 x200.a._super[17]._super._super,
                                 x200.a._super[18]._super._super,
                                 x200.a._super[19]._super._super,
                                 x200.a._super[20]._super._super,
                                 x200.a._super[21]._super._super,
                                 x200.a._super[22]._super._super,
                                 x200.a._super[23]._super._super,
                                 x200.a._super[24]._super._super,
                                 x200.a._super[25]._super._super,
                                 x200.a._super[26]._super._super,
                                 x200.a._super[27]._super._super,
                                 x200.a._super[28]._super._super,
                                 x200.a._super[29]._super._super,
                                 x200.a._super[30]._super._super,
                                 x200.a._super[31]._super._super,
                                 x201.a._super[0]._super._super,
                                 x201.a._super[1]._super._super,
                                 x201.a._super[2]._super._super,
                                 x201.a._super[3]._super._super,
                                 x201.a._super[4]._super._super,
                                 x201.a._super[5]._super._super,
                                 x201.a._super[6]._super._super,
                                 x201.a._super[7]._super._super,
                                 x201.a._super[8]._super._super,
                                 x201.a._super[9]._super._super,
                                 x201.a._super[10]._super._super,
                                 x201.a._super[11]._super._super,
                                 x201.a._super[12]._super._super,
                                 x201.a._super[13]._super._super,
                                 x201.a._super[14]._super._super,
                                 x201.a._super[15]._super._super,
                                 x201.a._super[16]._super._super,
                                 x201.a._super[17]._super._super,
                                 x201.a._super[18]._super._super,
                                 x201.a._super[19]._super._super,
                                 x201.a._super[20]._super._super,
                                 x201.a._super[21]._super._super,
                                 x201.a._super[22]._super._super,
                                 x201.a._super[23]._super._super,
                                 x201.a._super[24]._super._super,
                                 x201.a._super[25]._super._super,
                                 x201.a._super[26]._super._super,
                                 x201.a._super[27]._super._super,
                                 x201.a._super[28]._super._super,
                                 x201.a._super[29]._super._super,
                                 x201.a._super[30]._super._super,
                                 x201.a._super[31]._super._super,
                                 x194.e._super[0]._super._super,
                                 x194.e._super[1]._super._super,
                                 x194.e._super[2]._super._super,
                                 x194.e._super[3]._super._super,
                                 x194.e._super[4]._super._super,
                                 x194.e._super[5]._super._super,
                                 x194.e._super[6]._super._super,
                                 x194.e._super[7]._super._super,
                                 x194.e._super[8]._super._super,
                                 x194.e._super[9]._super._super,
                                 x194.e._super[10]._super._super,
                                 x194.e._super[11]._super._super,
                                 x194.e._super[12]._super._super,
                                 x194.e._super[13]._super._super,
                                 x194.e._super[14]._super._super,
                                 x194.e._super[15]._super._super,
                                 x194.e._super[16]._super._super,
                                 x194.e._super[17]._super._super,
                                 x194.e._super[18]._super._super,
                                 x194.e._super[19]._super._super,
                                 x194.e._super[20]._super._super,
                                 x194.e._super[21]._super._super,
                                 x194.e._super[22]._super._super,
                                 x194.e._super[23]._super._super,
                                 x194.e._super[24]._super._super,
                                 x194.e._super[25]._super._super,
                                 x194.e._super[26]._super._super,
                                 x194.e._super[27]._super._super,
                                 x194.e._super[28]._super._super,
                                 x194.e._super[29]._super._super,
                                 x194.e._super[30]._super._super,
                                 x194.e._super[31]._super._super,
                                 x195.e._super[0]._super._super,
                                 x195.e._super[1]._super._super,
                                 x195.e._super[2]._super._super,
                                 x195.e._super[3]._super._super,
                                 x195.e._super[4]._super._super,
                                 x195.e._super[5]._super._super,
                                 x195.e._super[6]._super._super,
                                 x195.e._super[7]._super._super,
                                 x195.e._super[8]._super._super,
                                 x195.e._super[9]._super._super,
                                 x195.e._super[10]._super._super,
                                 x195.e._super[11]._super._super,
                                 x195.e._super[12]._super._super,
                                 x195.e._super[13]._super._super,
                                 x195.e._super[14]._super._super,
                                 x195.e._super[15]._super._super,
                                 x195.e._super[16]._super._super,
                                 x195.e._super[17]._super._super,
                                 x195.e._super[18]._super._super,
                                 x195.e._super[19]._super._super,
                                 x195.e._super[20]._super._super,
                                 x195.e._super[21]._super._super,
                                 x195.e._super[22]._super._super,
                                 x195.e._super[23]._super._super,
                                 x195.e._super[24]._super._super,
                                 x195.e._super[25]._super._super,
                                 x195.e._super[26]._super._super,
                                 x195.e._super[27]._super._super,
                                 x195.e._super[28]._super._super,
                                 x195.e._super[29]._super._super,
                                 x195.e._super[30]._super._super,
                                 x195.e._super[31]._super._super,
                                 x196.e._super[0]._super._super,
                                 x196.e._super[1]._super._super,
                                 x196.e._super[2]._super._super,
                                 x196.e._super[3]._super._super,
                                 x196.e._super[4]._super._super,
                                 x196.e._super[5]._super._super,
                                 x196.e._super[6]._super._super,
                                 x196.e._super[7]._super._super,
                                 x196.e._super[8]._super._super,
                                 x196.e._super[9]._super._super,
                                 x196.e._super[10]._super._super,
                                 x196.e._super[11]._super._super,
                                 x196.e._super[12]._super._super,
                                 x196.e._super[13]._super._super,
                                 x196.e._super[14]._super._super,
                                 x196.e._super[15]._super._super,
                                 x196.e._super[16]._super._super,
                                 x196.e._super[17]._super._super,
                                 x196.e._super[18]._super._super,
                                 x196.e._super[19]._super._super,
                                 x196.e._super[20]._super._super,
                                 x196.e._super[21]._super._super,
                                 x196.e._super[22]._super._super,
                                 x196.e._super[23]._super._super,
                                 x196.e._super[24]._super._super,
                                 x196.e._super[25]._super._super,
                                 x196.e._super[26]._super._super,
                                 x196.e._super[27]._super._super,
                                 x196.e._super[28]._super._super,
                                 x196.e._super[29]._super._super,
                                 x196.e._super[30]._super._super,
                                 x196.e._super[31]._super._super,
                                 x197.e._super[0]._super._super,
                                 x197.e._super[1]._super._super,
                                 x197.e._super[2]._super._super,
                                 x197.e._super[3]._super._super,
                                 x197.e._super[4]._super._super,
                                 x197.e._super[5]._super._super,
                                 x197.e._super[6]._super._super,
                                 x197.e._super[7]._super._super,
                                 x197.e._super[8]._super._super,
                                 x197.e._super[9]._super._super,
                                 x197.e._super[10]._super._super,
                                 x197.e._super[11]._super._super,
                                 x197.e._super[12]._super._super,
                                 x197.e._super[13]._super._super,
                                 x197.e._super[14]._super._super,
                                 x197.e._super[15]._super._super,
                                 x197.e._super[16]._super._super,
                                 x197.e._super[17]._super._super,
                                 x197.e._super[18]._super._super,
                                 x197.e._super[19]._super._super,
                                 x197.e._super[20]._super._super,
                                 x197.e._super[21]._super._super,
                                 x197.e._super[22]._super._super,
                                 x197.e._super[23]._super._super,
                                 x197.e._super[24]._super._super,
                                 x197.e._super[25]._super._super,
                                 x197.e._super[26]._super._super,
                                 x197.e._super[27]._super._super,
                                 x197.e._super[28]._super._super,
                                 x197.e._super[29]._super._super,
                                 x197.e._super[30]._super._super,
                                 x197.e._super[31]._super._super,
                                 x198.e._super[0]._super._super,
                                 x198.e._super[1]._super._super,
                                 x198.e._super[2]._super._super,
                                 x198.e._super[3]._super._super,
                                 x198.e._super[4]._super._super,
                                 x198.e._super[5]._super._super,
                                 x198.e._super[6]._super._super,
                                 x198.e._super[7]._super._super,
                                 x198.e._super[8]._super._super,
                                 x198.e._super[9]._super._super,
                                 x198.e._super[10]._super._super,
                                 x198.e._super[11]._super._super,
                                 x198.e._super[12]._super._super,
                                 x198.e._super[13]._super._super,
                                 x198.e._super[14]._super._super,
                                 x198.e._super[15]._super._super,
                                 x198.e._super[16]._super._super,
                                 x198.e._super[17]._super._super,
                                 x198.e._super[18]._super._super,
                                 x198.e._super[19]._super._super,
                                 x198.e._super[20]._super._super,
                                 x198.e._super[21]._super._super,
                                 x198.e._super[22]._super._super,
                                 x198.e._super[23]._super._super,
                                 x198.e._super[24]._super._super,
                                 x198.e._super[25]._super._super,
                                 x198.e._super[26]._super._super,
                                 x198.e._super[27]._super._super,
                                 x198.e._super[28]._super._super,
                                 x198.e._super[29]._super._super,
                                 x198.e._super[30]._super._super,
                                 x198.e._super[31]._super._super,
                                 x199.e._super[0]._super._super,
                                 x199.e._super[1]._super._super,
                                 x199.e._super[2]._super._super,
                                 x199.e._super[3]._super._super,
                                 x199.e._super[4]._super._super,
                                 x199.e._super[5]._super._super,
                                 x199.e._super[6]._super._super,
                                 x199.e._super[7]._super._super,
                                 x199.e._super[8]._super._super,
                                 x199.e._super[9]._super._super,
                                 x199.e._super[10]._super._super,
                                 x199.e._super[11]._super._super,
                                 x199.e._super[12]._super._super,
                                 x199.e._super[13]._super._super,
                                 x199.e._super[14]._super._super,
                                 x199.e._super[15]._super._super,
                                 x199.e._super[16]._super._super,
                                 x199.e._super[17]._super._super,
                                 x199.e._super[18]._super._super,
                                 x199.e._super[19]._super._super,
                                 x199.e._super[20]._super._super,
                                 x199.e._super[21]._super._super,
                                 x199.e._super[22]._super._super,
                                 x199.e._super[23]._super._super,
                                 x199.e._super[24]._super._super,
                                 x199.e._super[25]._super._super,
                                 x199.e._super[26]._super._super,
                                 x199.e._super[27]._super._super,
                                 x199.e._super[28]._super._super,
                                 x199.e._super[29]._super._super,
                                 x199.e._super[30]._super._super,
                                 x199.e._super[31]._super._super,
                                 x200.e._super[0]._super._super,
                                 x200.e._super[1]._super._super,
                                 x200.e._super[2]._super._super,
                                 x200.e._super[3]._super._super,
                                 x200.e._super[4]._super._super,
                                 x200.e._super[5]._super._super,
                                 x200.e._super[6]._super._super,
                                 x200.e._super[7]._super._super,
                                 x200.e._super[8]._super._super,
                                 x200.e._super[9]._super._super,
                                 x200.e._super[10]._super._super,
                                 x200.e._super[11]._super._super,
                                 x200.e._super[12]._super._super,
                                 x200.e._super[13]._super._super,
                                 x200.e._super[14]._super._super,
                                 x200.e._super[15]._super._super,
                                 x200.e._super[16]._super._super,
                                 x200.e._super[17]._super._super,
                                 x200.e._super[18]._super._super,
                                 x200.e._super[19]._super._super,
                                 x200.e._super[20]._super._super,
                                 x200.e._super[21]._super._super,
                                 x200.e._super[22]._super._super,
                                 x200.e._super[23]._super._super,
                                 x200.e._super[24]._super._super,
                                 x200.e._super[25]._super._super,
                                 x200.e._super[26]._super._super,
                                 x200.e._super[27]._super._super,
                                 x200.e._super[28]._super._super,
                                 x200.e._super[29]._super._super,
                                 x200.e._super[30]._super._super,
                                 x200.e._super[31]._super._super,
                                 x201.e._super[0]._super._super,
                                 x201.e._super[1]._super._super,
                                 x201.e._super[2]._super._super,
                                 x201.e._super[3]._super._super,
                                 x201.e._super[4]._super._super,
                                 x201.e._super[5]._super._super,
                                 x201.e._super[6]._super._super,
                                 x201.e._super[7]._super._super,
                                 x201.e._super[8]._super._super,
                                 x201.e._super[9]._super._super,
                                 x201.e._super[10]._super._super,
                                 x201.e._super[11]._super._super,
                                 x201.e._super[12]._super._super,
                                 x201.e._super[13]._super._super,
                                 x201.e._super[14]._super._super,
                                 x201.e._super[15]._super._super,
                                 x201.e._super[16]._super._super,
                                 x201.e._super[17]._super._super,
                                 x201.e._super[18]._super._super,
                                 x201.e._super[19]._super._super,
                                 x201.e._super[20]._super._super,
                                 x201.e._super[21]._super._super,
                                 x201.e._super[22]._super._super,
                                 x201.e._super[23]._super._super,
                                 x201.e._super[24]._super._super,
                                 x201.e._super[25]._super._super,
                                 x201.e._super[26]._super._super,
                                 x201.e._super[27]._super._super,
                                 x201.e._super[28]._super._super,
                                 x201.e._super[29]._super._super,
                                 x201.e._super[30]._super._super,
                                 x201.e._super[31]._super._super,
                                 x194.w._super[0]._super._super,
                                 x194.w._super[1]._super._super,
                                 x194.w._super[2]._super._super,
                                 x194.w._super[3]._super._super,
                                 x194.w._super[4]._super._super,
                                 x194.w._super[5]._super._super,
                                 x194.w._super[6]._super._super,
                                 x194.w._super[7]._super._super,
                                 x194.w._super[8]._super._super,
                                 x194.w._super[9]._super._super,
                                 x194.w._super[10]._super._super,
                                 x194.w._super[11]._super._super,
                                 x194.w._super[12]._super._super,
                                 x194.w._super[13]._super._super,
                                 x194.w._super[14]._super._super,
                                 x194.w._super[15]._super._super,
                                 x194.w._super[16]._super._super,
                                 x194.w._super[17]._super._super,
                                 x194.w._super[18]._super._super,
                                 x194.w._super[19]._super._super,
                                 x194.w._super[20]._super._super,
                                 x194.w._super[21]._super._super,
                                 x194.w._super[22]._super._super,
                                 x194.w._super[23]._super._super,
                                 x194.w._super[24]._super._super,
                                 x194.w._super[25]._super._super,
                                 x194.w._super[26]._super._super,
                                 x194.w._super[27]._super._super,
                                 x194.w._super[28]._super._super,
                                 x194.w._super[29]._super._super,
                                 x194.w._super[30]._super._super,
                                 x194.w._super[31]._super._super,
                                 x195.w._super[0]._super._super,
                                 x195.w._super[1]._super._super,
                                 x195.w._super[2]._super._super,
                                 x195.w._super[3]._super._super,
                                 x195.w._super[4]._super._super,
                                 x195.w._super[5]._super._super,
                                 x195.w._super[6]._super._super,
                                 x195.w._super[7]._super._super,
                                 x195.w._super[8]._super._super,
                                 x195.w._super[9]._super._super,
                                 x195.w._super[10]._super._super,
                                 x195.w._super[11]._super._super,
                                 x195.w._super[12]._super._super,
                                 x195.w._super[13]._super._super,
                                 x195.w._super[14]._super._super,
                                 x195.w._super[15]._super._super,
                                 x195.w._super[16]._super._super,
                                 x195.w._super[17]._super._super,
                                 x195.w._super[18]._super._super,
                                 x195.w._super[19]._super._super,
                                 x195.w._super[20]._super._super,
                                 x195.w._super[21]._super._super,
                                 x195.w._super[22]._super._super,
                                 x195.w._super[23]._super._super,
                                 x195.w._super[24]._super._super,
                                 x195.w._super[25]._super._super,
                                 x195.w._super[26]._super._super,
                                 x195.w._super[27]._super._super,
                                 x195.w._super[28]._super._super,
                                 x195.w._super[29]._super._super,
                                 x195.w._super[30]._super._super,
                                 x195.w._super[31]._super._super,
                                 x196.w._super[0]._super._super,
                                 x196.w._super[1]._super._super,
                                 x196.w._super[2]._super._super,
                                 x196.w._super[3]._super._super,
                                 x196.w._super[4]._super._super,
                                 x196.w._super[5]._super._super,
                                 x196.w._super[6]._super._super,
                                 x196.w._super[7]._super._super,
                                 x196.w._super[8]._super._super,
                                 x196.w._super[9]._super._super,
                                 x196.w._super[10]._super._super,
                                 x196.w._super[11]._super._super,
                                 x196.w._super[12]._super._super,
                                 x196.w._super[13]._super._super,
                                 x196.w._super[14]._super._super,
                                 x196.w._super[15]._super._super,
                                 x196.w._super[16]._super._super,
                                 x196.w._super[17]._super._super,
                                 x196.w._super[18]._super._super,
                                 x196.w._super[19]._super._super,
                                 x196.w._super[20]._super._super,
                                 x196.w._super[21]._super._super,
                                 x196.w._super[22]._super._super,
                                 x196.w._super[23]._super._super,
                                 x196.w._super[24]._super._super,
                                 x196.w._super[25]._super._super,
                                 x196.w._super[26]._super._super,
                                 x196.w._super[27]._super._super,
                                 x196.w._super[28]._super._super,
                                 x196.w._super[29]._super._super,
                                 x196.w._super[30]._super._super,
                                 x196.w._super[31]._super._super,
                                 x197.w._super[0]._super._super,
                                 x197.w._super[1]._super._super,
                                 x197.w._super[2]._super._super,
                                 x197.w._super[3]._super._super,
                                 x197.w._super[4]._super._super,
                                 x197.w._super[5]._super._super,
                                 x197.w._super[6]._super._super,
                                 x197.w._super[7]._super._super,
                                 x197.w._super[8]._super._super,
                                 x197.w._super[9]._super._super,
                                 x197.w._super[10]._super._super,
                                 x197.w._super[11]._super._super,
                                 x197.w._super[12]._super._super,
                                 x197.w._super[13]._super._super,
                                 x197.w._super[14]._super._super,
                                 x197.w._super[15]._super._super,
                                 x197.w._super[16]._super._super,
                                 x197.w._super[17]._super._super,
                                 x197.w._super[18]._super._super,
                                 x197.w._super[19]._super._super,
                                 x197.w._super[20]._super._super,
                                 x197.w._super[21]._super._super,
                                 x197.w._super[22]._super._super,
                                 x197.w._super[23]._super._super,
                                 x197.w._super[24]._super._super,
                                 x197.w._super[25]._super._super,
                                 x197.w._super[26]._super._super,
                                 x197.w._super[27]._super._super,
                                 x197.w._super[28]._super._super,
                                 x197.w._super[29]._super._super,
                                 x197.w._super[30]._super._super,
                                 x197.w._super[31]._super._super,
                                 x198.w._super[0]._super._super,
                                 x198.w._super[1]._super._super,
                                 x198.w._super[2]._super._super,
                                 x198.w._super[3]._super._super,
                                 x198.w._super[4]._super._super,
                                 x198.w._super[5]._super._super,
                                 x198.w._super[6]._super._super,
                                 x198.w._super[7]._super._super,
                                 x198.w._super[8]._super._super,
                                 x198.w._super[9]._super._super,
                                 x198.w._super[10]._super._super,
                                 x198.w._super[11]._super._super,
                                 x198.w._super[12]._super._super,
                                 x198.w._super[13]._super._super,
                                 x198.w._super[14]._super._super,
                                 x198.w._super[15]._super._super,
                                 x198.w._super[16]._super._super,
                                 x198.w._super[17]._super._super,
                                 x198.w._super[18]._super._super,
                                 x198.w._super[19]._super._super,
                                 x198.w._super[20]._super._super,
                                 x198.w._super[21]._super._super,
                                 x198.w._super[22]._super._super,
                                 x198.w._super[23]._super._super,
                                 x198.w._super[24]._super._super,
                                 x198.w._super[25]._super._super,
                                 x198.w._super[26]._super._super,
                                 x198.w._super[27]._super._super,
                                 x198.w._super[28]._super._super,
                                 x198.w._super[29]._super._super,
                                 x198.w._super[30]._super._super,
                                 x198.w._super[31]._super._super,
                                 x199.w._super[0]._super._super,
                                 x199.w._super[1]._super._super,
                                 x199.w._super[2]._super._super,
                                 x199.w._super[3]._super._super,
                                 x199.w._super[4]._super._super,
                                 x199.w._super[5]._super._super,
                                 x199.w._super[6]._super._super,
                                 x199.w._super[7]._super._super,
                                 x199.w._super[8]._super._super,
                                 x199.w._super[9]._super._super,
                                 x199.w._super[10]._super._super,
                                 x199.w._super[11]._super._super,
                                 x199.w._super[12]._super._super,
                                 x199.w._super[13]._super._super,
                                 x199.w._super[14]._super._super,
                                 x199.w._super[15]._super._super,
                                 x199.w._super[16]._super._super,
                                 x199.w._super[17]._super._super,
                                 x199.w._super[18]._super._super,
                                 x199.w._super[19]._super._super,
                                 x199.w._super[20]._super._super,
                                 x199.w._super[21]._super._super,
                                 x199.w._super[22]._super._super,
                                 x199.w._super[23]._super._super,
                                 x199.w._super[24]._super._super,
                                 x199.w._super[25]._super._super,
                                 x199.w._super[26]._super._super,
                                 x199.w._super[27]._super._super,
                                 x199.w._super[28]._super._super,
                                 x199.w._super[29]._super._super,
                                 x199.w._super[30]._super._super,
                                 x199.w._super[31]._super._super,
                                 x200.w._super[0]._super._super,
                                 x200.w._super[1]._super._super,
                                 x200.w._super[2]._super._super,
                                 x200.w._super[3]._super._super,
                                 x200.w._super[4]._super._super,
                                 x200.w._super[5]._super._super,
                                 x200.w._super[6]._super._super,
                                 x200.w._super[7]._super._super,
                                 x200.w._super[8]._super._super,
                                 x200.w._super[9]._super._super,
                                 x200.w._super[10]._super._super,
                                 x200.w._super[11]._super._super,
                                 x200.w._super[12]._super._super,
                                 x200.w._super[13]._super._super,
                                 x200.w._super[14]._super._super,
                                 x200.w._super[15]._super._super,
                                 x200.w._super[16]._super._super,
                                 x200.w._super[17]._super._super,
                                 x200.w._super[18]._super._super,
                                 x200.w._super[19]._super._super,
                                 x200.w._super[20]._super._super,
                                 x200.w._super[21]._super._super,
                                 x200.w._super[22]._super._super,
                                 x200.w._super[23]._super._super,
                                 x200.w._super[24]._super._super,
                                 x200.w._super[25]._super._super,
                                 x200.w._super[26]._super._super,
                                 x200.w._super[27]._super._super,
                                 x200.w._super[28]._super._super,
                                 x200.w._super[29]._super._super,
                                 x200.w._super[30]._super._super,
                                 x200.w._super[31]._super._super,
                                 x201.w._super[0]._super._super,
                                 x201.w._super[1]._super._super,
                                 x201.w._super[2]._super._super,
                                 x201.w._super[3]._super._super,
                                 x201.w._super[4]._super._super,
                                 x201.w._super[5]._super._super,
                                 x201.w._super[6]._super._super,
                                 x201.w._super[7]._super._super,
                                 x201.w._super[8]._super._super,
                                 x201.w._super[9]._super._super,
                                 x201.w._super[10]._super._super,
                                 x201.w._super[11]._super._super,
                                 x201.w._super[12]._super._super,
                                 x201.w._super[13]._super._super,
                                 x201.w._super[14]._super._super,
                                 x201.w._super[15]._super._super,
                                 x201.w._super[16]._super._super,
                                 x201.w._super[17]._super._super,
                                 x201.w._super[18]._super._super,
                                 x201.w._super[19]._super._super,
                                 x201.w._super[20]._super._super,
                                 x201.w._super[21]._super._super,
                                 x201.w._super[22]._super._super,
                                 x201.w._super[23]._super._super,
                                 x201.w._super[24]._super._super,
                                 x201.w._super[25]._super._super,
                                 x201.w._super[26]._super._super,
                                 x201.w._super[27]._super._super,
                                 x201.w._super[28]._super._super,
                                 x201.w._super[29]._super._super,
                                 x201.w._super[30]._super._super,
                                 x201.w._super[31]._super._super,
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0)};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:186)
  Val16Array x203 = Val16Array{arg0.sflat[0]._super,
                               arg0.sflat[1]._super,
                               arg0.sflat[2]._super,
                               arg0.sflat[3]._super,
                               arg0.sflat[4]._super,
                               arg0.sflat[5]._super,
                               arg0.sflat[6]._super,
                               arg0.sflat[7]._super,
                               arg0.sflat[8]._super,
                               arg0.sflat[9]._super,
                               arg0.sflat[10]._super,
                               arg0.sflat[11]._super,
                               arg0.sflat[12]._super,
                               arg0.sflat[13]._super,
                               arg0.sflat[14]._super,
                               arg0.sflat[15]._super};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:208)
  TopStateStruct x204 = exec_TopState(ctx, x202, x192, x203, LAYOUT_LOOKUP(layout4, _super));
  return x204;
}
__device__ OneHot_12_Struct exec_OneHot_12_(ExecContext& ctx,
                                            Val arg0,
                                            BoundLayout<OneHot_12_Layout> layout1) {
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:7)
  OneHot_12__SuperStruct12Array x2 = map(
      Val12Array{Val(0),
                 Val(1),
                 Val(2),
                 Val(3),
                 Val(4),
                 Val(5),
                 Val(6),
                 Val(7),
                 Val(8),
                 Val(9),
                 Val(10),
                 Val(11)},
      LAYOUT_LOOKUP(layout1, _super),
      ([&](Val12Array::value_type x3, BoundLayout<NondetRegLayout12LayoutArray::value_type> x4) {
        NondetRegStruct x5 = exec_NondetBitReg(ctx, isz((x3 - arg0)), x4);
        return OneHot_12__SuperStruct{._super = x5};
      }));
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:9)
  Val x6 = (x2[0]._super._super + x2[1]._super._super);
  Val x7 = ((x6 + x2[2]._super._super) + x2[3]._super._super);
  Val x8 = ((x7 + x2[4]._super._super) + x2[5]._super._super);
  Val x9 = ((x8 + x2[6]._super._super) + x2[7]._super._super);
  Val x10 = ((x9 + x2[8]._super._super) + x2[9]._super._super);
  Val x11 = ((x10 + x2[10]._super._super) + x2[11]._super._super);
  EQZ((x11 - Val(1)), "OneHot(zirgen/circuit/keccak2/one_hot.zir:9)");
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:11)
  Val x12 = (x2[2]._super._super * Val(2));
  Val x13 = (x2[3]._super._super * Val(3));
  Val x14 = (x2[4]._super._super * Val(4));
  Val x15 = (x2[5]._super._super * Val(5));
  Val x16 = (x2[6]._super._super * Val(6));
  Val x17 = (x2[7]._super._super * Val(7));
  Val x18 = (x2[8]._super._super * Val(8));
  Val x19 = (x2[9]._super._super * Val(9));
  Val x20 = (x2[10]._super._super * Val(10));
  Val x21 = (x2[11]._super._super * Val(11));
  Val x22 = (x2[1]._super._super + x12);
  Val x23 = (((x22 + x13) + x14) + x15);
  Val x24 = (((x23 + x16) + x17) + x18);
  Val x25 = (((x24 + x19) + x20) + x21);
  EQZ((x25 - arg0), "OneHot(zirgen/circuit/keccak2/one_hot.zir:11)");
  return OneHot_12_Struct{.bits = x2};
}

} // namespace risc0::circuit::keccak::cuda
