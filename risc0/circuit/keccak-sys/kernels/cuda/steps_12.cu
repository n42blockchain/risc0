#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"

namespace risc0::impl {

__device__ TopStateStruct exec_ShaCycle(ExecContext& ctx,
                                        TopStateStruct arg0,
                                        TopStateStruct arg1,
                                        Val arg2,
                                        Val arg3,
                                        BoundLayout<ShaCycleLayout> layout4) {
  // Log(<preamble>:22)
  // ShaCycle(zirgen/circuit/keccak2/top.zir:187)
  INVOKE_EXTERN(ctx, log, "ShaCycle", std::initializer_list<Val>{arg3, arg2});
  // LoadShaState(zirgen/circuit/keccak2/top.zir:147)
  // ShaCycle(zirgen/circuit/keccak2/top.zir:189)
  TopStateBitsStruct800Array x5 = arg0.bits;
  TopStateBitsStruct800Array x6 = arg1.bits;
  // LoadShaState(zirgen/circuit/keccak2/top.zir:148)
  Val x7 = x5[224]._super._super;
  Val x8 = x5[225]._super._super;
  Val x9 = x5[226]._super._super;
  Val x10 = x5[227]._super._super;
  Val x11 = x5[228]._super._super;
  Val x12 = x5[229]._super._super;
  Val x13 = x5[230]._super._super;
  Val x14 = x5[231]._super._super;
  Val x15 = x5[232]._super._super;
  Val x16 = x5[233]._super._super;
  Val x17 = x5[234]._super._super;
  Val x18 = x5[235]._super._super;
  Val x19 = x5[236]._super._super;
  Val x20 = x5[237]._super._super;
  Val x21 = x5[238]._super._super;
  Val x22 = x5[239]._super._super;
  Val x23 = x5[240]._super._super;
  Val x24 = x5[241]._super._super;
  Val x25 = x5[242]._super._super;
  Val x26 = x5[243]._super._super;
  Val x27 = x5[244]._super._super;
  Val x28 = x5[245]._super._super;
  Val x29 = x5[246]._super._super;
  Val x30 = x5[247]._super._super;
  Val x31 = x5[248]._super._super;
  Val x32 = x5[249]._super._super;
  Val x33 = x5[250]._super._super;
  Val x34 = x5[251]._super._super;
  Val x35 = x5[252]._super._super;
  Val x36 = x5[253]._super._super;
  Val x37 = x5[254]._super._super;
  Val x38 = x5[255]._super._super;
  Val x39 = x5[192]._super._super;
  Val x40 = x5[193]._super._super;
  Val x41 = x5[194]._super._super;
  Val x42 = x5[195]._super._super;
  Val x43 = x5[196]._super._super;
  Val x44 = x5[197]._super._super;
  Val x45 = x5[198]._super._super;
  Val x46 = x5[199]._super._super;
  Val x47 = x5[200]._super._super;
  Val x48 = x5[201]._super._super;
  Val x49 = x5[202]._super._super;
  Val x50 = x5[203]._super._super;
  Val x51 = x5[204]._super._super;
  Val x52 = x5[205]._super._super;
  Val x53 = x5[206]._super._super;
  Val x54 = x5[207]._super._super;
  Val x55 = x5[208]._super._super;
  Val x56 = x5[209]._super._super;
  Val x57 = x5[210]._super._super;
  Val x58 = x5[211]._super._super;
  Val x59 = x5[212]._super._super;
  Val x60 = x5[213]._super._super;
  Val x61 = x5[214]._super._super;
  Val x62 = x5[215]._super._super;
  Val x63 = x5[216]._super._super;
  Val x64 = x5[217]._super._super;
  Val x65 = x5[218]._super._super;
  Val x66 = x5[219]._super._super;
  Val x67 = x5[220]._super._super;
  Val x68 = x5[221]._super._super;
  Val x69 = x5[222]._super._super;
  Val x70 = x5[223]._super._super;
  Val x71 = x5[160]._super._super;
  Val x72 = x5[161]._super._super;
  Val x73 = x5[162]._super._super;
  Val x74 = x5[163]._super._super;
  Val x75 = x5[164]._super._super;
  Val x76 = x5[165]._super._super;
  Val x77 = x5[166]._super._super;
  Val x78 = x5[167]._super._super;
  Val x79 = x5[168]._super._super;
  Val x80 = x5[169]._super._super;
  Val x81 = x5[170]._super._super;
  Val x82 = x5[171]._super._super;
  Val x83 = x5[172]._super._super;
  Val x84 = x5[173]._super._super;
  Val x85 = x5[174]._super._super;
  Val x86 = x5[175]._super._super;
  Val x87 = x5[176]._super._super;
  Val x88 = x5[177]._super._super;
  Val x89 = x5[178]._super._super;
  Val x90 = x5[179]._super._super;
  Val x91 = x5[180]._super._super;
  Val x92 = x5[181]._super._super;
  Val x93 = x5[182]._super._super;
  Val x94 = x5[183]._super._super;
  Val x95 = x5[184]._super._super;
  Val x96 = x5[185]._super._super;
  Val x97 = x5[186]._super._super;
  Val x98 = x5[187]._super._super;
  Val x99 = x5[188]._super._super;
  Val x100 = x5[189]._super._super;
  Val x101 = x5[190]._super._super;
  Val x102 = x5[191]._super._super;
  Val x103 = x5[128]._super._super;
  Val x104 = x5[129]._super._super;
  Val x105 = x5[130]._super._super;
  Val x106 = x5[131]._super._super;
  Val x107 = x5[132]._super._super;
  Val x108 = x5[133]._super._super;
  Val x109 = x5[134]._super._super;
  Val x110 = x5[135]._super._super;
  Val x111 = x5[136]._super._super;
  Val x112 = x5[137]._super._super;
  Val x113 = x5[138]._super._super;
  Val x114 = x5[139]._super._super;
  Val x115 = x5[140]._super._super;
  Val x116 = x5[141]._super._super;
  Val x117 = x5[142]._super._super;
  Val x118 = x5[143]._super._super;
  Val x119 = x5[144]._super._super;
  Val x120 = x5[145]._super._super;
  Val x121 = x5[146]._super._super;
  Val x122 = x5[147]._super._super;
  Val x123 = x5[148]._super._super;
  Val x124 = x5[149]._super._super;
  Val x125 = x5[150]._super._super;
  Val x126 = x5[151]._super._super;
  Val x127 = x5[152]._super._super;
  Val x128 = x5[153]._super._super;
  Val x129 = x5[154]._super._super;
  Val x130 = x5[155]._super._super;
  Val x131 = x5[156]._super._super;
  Val x132 = x5[157]._super._super;
  Val x133 = x5[158]._super._super;
  Val x134 = x5[159]._super._super;
  // LoadShaState(zirgen/circuit/keccak2/top.zir:149)
  Val x135 = x5[480]._super._super;
  Val x136 = x5[481]._super._super;
  Val x137 = x5[482]._super._super;
  Val x138 = x5[483]._super._super;
  Val x139 = x5[484]._super._super;
  Val x140 = x5[485]._super._super;
  Val x141 = x5[486]._super._super;
  Val x142 = x5[487]._super._super;
  Val x143 = x5[488]._super._super;
  Val x144 = x5[489]._super._super;
  Val x145 = x5[490]._super._super;
  Val x146 = x5[491]._super._super;
  Val x147 = x5[492]._super._super;
  Val x148 = x5[493]._super._super;
  Val x149 = x5[494]._super._super;
  Val x150 = x5[495]._super._super;
  Val x151 = x5[496]._super._super;
  Val x152 = x5[497]._super._super;
  Val x153 = x5[498]._super._super;
  Val x154 = x5[499]._super._super;
  Val x155 = x5[500]._super._super;
  Val x156 = x5[501]._super._super;
  Val x157 = x5[502]._super._super;
  Val x158 = x5[503]._super._super;
  Val x159 = x5[504]._super._super;
  Val x160 = x5[505]._super._super;
  Val x161 = x5[506]._super._super;
  Val x162 = x5[507]._super._super;
  Val x163 = x5[508]._super._super;
  Val x164 = x5[509]._super._super;
  Val x165 = x5[510]._super._super;
  Val x166 = x5[511]._super._super;
  Val x167 = x5[448]._super._super;
  Val x168 = x5[449]._super._super;
  Val x169 = x5[450]._super._super;
  Val x170 = x5[451]._super._super;
  Val x171 = x5[452]._super._super;
  Val x172 = x5[453]._super._super;
  Val x173 = x5[454]._super._super;
  Val x174 = x5[455]._super._super;
  Val x175 = x5[456]._super._super;
  Val x176 = x5[457]._super._super;
  Val x177 = x5[458]._super._super;
  Val x178 = x5[459]._super._super;
  Val x179 = x5[460]._super._super;
  Val x180 = x5[461]._super._super;
  Val x181 = x5[462]._super._super;
  Val x182 = x5[463]._super._super;
  Val x183 = x5[464]._super._super;
  Val x184 = x5[465]._super._super;
  Val x185 = x5[466]._super._super;
  Val x186 = x5[467]._super._super;
  Val x187 = x5[468]._super._super;
  Val x188 = x5[469]._super._super;
  Val x189 = x5[470]._super._super;
  Val x190 = x5[471]._super._super;
  Val x191 = x5[472]._super._super;
  Val x192 = x5[473]._super._super;
  Val x193 = x5[474]._super._super;
  Val x194 = x5[475]._super._super;
  Val x195 = x5[476]._super._super;
  Val x196 = x5[477]._super._super;
  Val x197 = x5[478]._super._super;
  Val x198 = x5[479]._super._super;
  Val x199 = x5[416]._super._super;
  Val x200 = x5[417]._super._super;
  Val x201 = x5[418]._super._super;
  Val x202 = x5[419]._super._super;
  Val x203 = x5[420]._super._super;
  Val x204 = x5[421]._super._super;
  Val x205 = x5[422]._super._super;
  Val x206 = x5[423]._super._super;
  Val x207 = x5[424]._super._super;
  Val x208 = x5[425]._super._super;
  Val x209 = x5[426]._super._super;
  Val x210 = x5[427]._super._super;
  Val x211 = x5[428]._super._super;
  Val x212 = x5[429]._super._super;
  Val x213 = x5[430]._super._super;
  Val x214 = x5[431]._super._super;
  Val x215 = x5[432]._super._super;
  Val x216 = x5[433]._super._super;
  Val x217 = x5[434]._super._super;
  Val x218 = x5[435]._super._super;
  Val x219 = x5[436]._super._super;
  Val x220 = x5[437]._super._super;
  Val x221 = x5[438]._super._super;
  Val x222 = x5[439]._super._super;
  Val x223 = x5[440]._super._super;
  Val x224 = x5[441]._super._super;
  Val x225 = x5[442]._super._super;
  Val x226 = x5[443]._super._super;
  Val x227 = x5[444]._super._super;
  Val x228 = x5[445]._super._super;
  Val x229 = x5[446]._super._super;
  Val x230 = x5[447]._super._super;
  Val x231 = x5[384]._super._super;
  Val x232 = x5[385]._super._super;
  Val x233 = x5[386]._super._super;
  Val x234 = x5[387]._super._super;
  Val x235 = x5[388]._super._super;
  Val x236 = x5[389]._super._super;
  Val x237 = x5[390]._super._super;
  Val x238 = x5[391]._super._super;
  Val x239 = x5[392]._super._super;
  Val x240 = x5[393]._super._super;
  Val x241 = x5[394]._super._super;
  Val x242 = x5[395]._super._super;
  Val x243 = x5[396]._super._super;
  Val x244 = x5[397]._super._super;
  Val x245 = x5[398]._super._super;
  Val x246 = x5[399]._super._super;
  Val x247 = x5[400]._super._super;
  Val x248 = x5[401]._super._super;
  Val x249 = x5[402]._super._super;
  Val x250 = x5[403]._super._super;
  Val x251 = x5[404]._super._super;
  Val x252 = x5[405]._super._super;
  Val x253 = x5[406]._super._super;
  Val x254 = x5[407]._super._super;
  Val x255 = x5[408]._super._super;
  Val x256 = x5[409]._super._super;
  Val x257 = x5[410]._super._super;
  Val x258 = x5[411]._super._super;
  Val x259 = x5[412]._super._super;
  Val x260 = x5[413]._super._super;
  Val x261 = x5[414]._super._super;
  Val x262 = x5[415]._super._super;
  // LoadShaState(zirgen/circuit/keccak2/top.zir:150)
  Val x263 = x5[736]._super._super;
  Val x264 = x5[737]._super._super;
  Val x265 = x5[738]._super._super;
  Val x266 = x5[739]._super._super;
  Val x267 = x5[740]._super._super;
  Val x268 = x5[741]._super._super;
  Val x269 = x5[742]._super._super;
  Val x270 = x5[743]._super._super;
  Val x271 = x5[744]._super._super;
  Val x272 = x5[745]._super._super;
  Val x273 = x5[746]._super._super;
  Val x274 = x5[747]._super._super;
  Val x275 = x5[748]._super._super;
  Val x276 = x5[749]._super._super;
  Val x277 = x5[750]._super._super;
  Val x278 = x5[751]._super._super;
  Val x279 = x5[752]._super._super;
  Val x280 = x5[753]._super._super;
  Val x281 = x5[754]._super._super;
  Val x282 = x5[755]._super._super;
  Val x283 = x5[756]._super._super;
  Val x284 = x5[757]._super._super;
  Val x285 = x5[758]._super._super;
  Val x286 = x5[759]._super._super;
  Val x287 = x5[760]._super._super;
  Val x288 = x5[761]._super._super;
  Val x289 = x5[762]._super._super;
  Val x290 = x5[763]._super._super;
  Val x291 = x5[764]._super._super;
  Val x292 = x5[765]._super._super;
  Val x293 = x5[766]._super._super;
  Val x294 = x5[767]._super._super;
  Val x295 = x5[704]._super._super;
  Val x296 = x5[705]._super._super;
  Val x297 = x5[706]._super._super;
  Val x298 = x5[707]._super._super;
  Val x299 = x5[708]._super._super;
  Val x300 = x5[709]._super._super;
  Val x301 = x5[710]._super._super;
  Val x302 = x5[711]._super._super;
  Val x303 = x5[712]._super._super;
  Val x304 = x5[713]._super._super;
  Val x305 = x5[714]._super._super;
  Val x306 = x5[715]._super._super;
  Val x307 = x5[716]._super._super;
  Val x308 = x5[717]._super._super;
  Val x309 = x5[718]._super._super;
  Val x310 = x5[719]._super._super;
  Val x311 = x5[720]._super._super;
  Val x312 = x5[721]._super._super;
  Val x313 = x5[722]._super._super;
  Val x314 = x5[723]._super._super;
  Val x315 = x5[724]._super._super;
  Val x316 = x5[725]._super._super;
  Val x317 = x5[726]._super._super;
  Val x318 = x5[727]._super._super;
  Val x319 = x5[728]._super._super;
  Val x320 = x5[729]._super._super;
  Val x321 = x5[730]._super._super;
  Val x322 = x5[731]._super._super;
  Val x323 = x5[732]._super._super;
  Val x324 = x5[733]._super._super;
  Val x325 = x5[734]._super._super;
  Val x326 = x5[735]._super._super;
  Val x327 = x5[672]._super._super;
  Val x328 = x5[673]._super._super;
  Val x329 = x5[674]._super._super;
  Val x330 = x5[675]._super._super;
  Val x331 = x5[676]._super._super;
  Val x332 = x5[677]._super._super;
  Val x333 = x5[678]._super._super;
  Val x334 = x5[679]._super._super;
  Val x335 = x5[680]._super._super;
  Val x336 = x5[681]._super._super;
  Val x337 = x5[682]._super._super;
  Val x338 = x5[683]._super._super;
  Val x339 = x5[684]._super._super;
  Val x340 = x5[685]._super._super;
  Val x341 = x5[686]._super._super;
  Val x342 = x5[687]._super._super;
  Val x343 = x5[688]._super._super;
  Val x344 = x5[689]._super._super;
  Val x345 = x5[690]._super._super;
  Val x346 = x5[691]._super._super;
  Val x347 = x5[692]._super._super;
  Val x348 = x5[693]._super._super;
  Val x349 = x5[694]._super._super;
  Val x350 = x5[695]._super._super;
  Val x351 = x5[696]._super._super;
  Val x352 = x5[697]._super._super;
  Val x353 = x5[698]._super._super;
  Val x354 = x5[699]._super._super;
  Val x355 = x5[700]._super._super;
  Val x356 = x5[701]._super._super;
  Val x357 = x5[702]._super._super;
  Val x358 = x5[703]._super._super;
  Val x359 = x5[640]._super._super;
  Val x360 = x5[641]._super._super;
  Val x361 = x5[642]._super._super;
  Val x362 = x5[643]._super._super;
  Val x363 = x5[644]._super._super;
  Val x364 = x5[645]._super._super;
  Val x365 = x5[646]._super._super;
  Val x366 = x5[647]._super._super;
  Val x367 = x5[648]._super._super;
  Val x368 = x5[649]._super._super;
  Val x369 = x5[650]._super._super;
  Val x370 = x5[651]._super._super;
  Val x371 = x5[652]._super._super;
  Val x372 = x5[653]._super._super;
  Val x373 = x5[654]._super._super;
  Val x374 = x5[655]._super._super;
  Val x375 = x5[656]._super._super;
  Val x376 = x5[657]._super._super;
  Val x377 = x5[658]._super._super;
  Val x378 = x5[659]._super._super;
  Val x379 = x5[660]._super._super;
  Val x380 = x5[661]._super._super;
  Val x381 = x5[662]._super._super;
  Val x382 = x5[663]._super._super;
  Val x383 = x5[664]._super._super;
  Val x384 = x5[665]._super._super;
  Val x385 = x5[666]._super._super;
  Val x386 = x5[667]._super._super;
  Val x387 = x5[668]._super._super;
  Val x388 = x5[669]._super._super;
  Val x389 = x5[670]._super._super;
  Val x390 = x5[671]._super._super;
  Val x391 = x5[608]._super._super;
  Val x392 = x5[609]._super._super;
  Val x393 = x5[610]._super._super;
  Val x394 = x5[611]._super._super;
  Val x395 = x5[612]._super._super;
  Val x396 = x5[613]._super._super;
  Val x397 = x5[614]._super._super;
  Val x398 = x5[615]._super._super;
  Val x399 = x5[616]._super._super;
  Val x400 = x5[617]._super._super;
  Val x401 = x5[618]._super._super;
  Val x402 = x5[619]._super._super;
  Val x403 = x5[620]._super._super;
  Val x404 = x5[621]._super._super;
  Val x405 = x5[622]._super._super;
  Val x406 = x5[623]._super._super;
  Val x407 = x5[624]._super._super;
  Val x408 = x5[625]._super._super;
  Val x409 = x5[626]._super._super;
  Val x410 = x5[627]._super._super;
  Val x411 = x5[628]._super._super;
  Val x412 = x5[629]._super._super;
  Val x413 = x5[630]._super._super;
  Val x414 = x5[631]._super._super;
  Val x415 = x5[632]._super._super;
  Val x416 = x5[633]._super._super;
  Val x417 = x5[634]._super._super;
  Val x418 = x5[635]._super._super;
  Val x419 = x5[636]._super._super;
  Val x420 = x5[637]._super._super;
  Val x421 = x5[638]._super._super;
  Val x422 = x5[639]._super._super;
  Val x423 = x5[576]._super._super;
  Val x424 = x5[577]._super._super;
  Val x425 = x5[578]._super._super;
  Val x426 = x5[579]._super._super;
  Val x427 = x5[580]._super._super;
  Val x428 = x5[581]._super._super;
  Val x429 = x5[582]._super._super;
  Val x430 = x5[583]._super._super;
  Val x431 = x5[584]._super._super;
  Val x432 = x5[585]._super._super;
  Val x433 = x5[586]._super._super;
  Val x434 = x5[587]._super._super;
  Val x435 = x5[588]._super._super;
  Val x436 = x5[589]._super._super;
  Val x437 = x5[590]._super._super;
  Val x438 = x5[591]._super._super;
  Val x439 = x5[592]._super._super;
  Val x440 = x5[593]._super._super;
  Val x441 = x5[594]._super._super;
  Val x442 = x5[595]._super._super;
  Val x443 = x5[596]._super._super;
  Val x444 = x5[597]._super._super;
  Val x445 = x5[598]._super._super;
  Val x446 = x5[599]._super._super;
  Val x447 = x5[600]._super._super;
  Val x448 = x5[601]._super._super;
  Val x449 = x5[602]._super._super;
  Val x450 = x5[603]._super._super;
  Val x451 = x5[604]._super._super;
  Val x452 = x5[605]._super._super;
  Val x453 = x5[606]._super._super;
  Val x454 = x5[607]._super._super;
  Val x455 = x5[544]._super._super;
  Val x456 = x5[545]._super._super;
  Val x457 = x5[546]._super._super;
  Val x458 = x5[547]._super._super;
  Val x459 = x5[548]._super._super;
  Val x460 = x5[549]._super._super;
  Val x461 = x5[550]._super._super;
  Val x462 = x5[551]._super._super;
  Val x463 = x5[552]._super._super;
  Val x464 = x5[553]._super._super;
  Val x465 = x5[554]._super._super;
  Val x466 = x5[555]._super._super;
  Val x467 = x5[556]._super._super;
  Val x468 = x5[557]._super._super;
  Val x469 = x5[558]._super._super;
  Val x470 = x5[559]._super._super;
  Val x471 = x5[560]._super._super;
  Val x472 = x5[561]._super._super;
  Val x473 = x5[562]._super._super;
  Val x474 = x5[563]._super._super;
  Val x475 = x5[564]._super._super;
  Val x476 = x5[565]._super._super;
  Val x477 = x5[566]._super._super;
  Val x478 = x5[567]._super._super;
  Val x479 = x5[568]._super._super;
  Val x480 = x5[569]._super._super;
  Val x481 = x5[570]._super._super;
  Val x482 = x5[571]._super._super;
  Val x483 = x5[572]._super._super;
  Val x484 = x5[573]._super._super;
  Val x485 = x5[574]._super._super;
  Val x486 = x5[575]._super._super;
  Val x487 = x5[512]._super._super;
  Val x488 = x5[513]._super._super;
  Val x489 = x5[514]._super._super;
  Val x490 = x5[515]._super._super;
  Val x491 = x5[516]._super._super;
  Val x492 = x5[517]._super._super;
  Val x493 = x5[518]._super._super;
  Val x494 = x5[519]._super._super;
  Val x495 = x5[520]._super._super;
  Val x496 = x5[521]._super._super;
  Val x497 = x5[522]._super._super;
  Val x498 = x5[523]._super._super;
  Val x499 = x5[524]._super._super;
  Val x500 = x5[525]._super._super;
  Val x501 = x5[526]._super._super;
  Val x502 = x5[527]._super._super;
  Val x503 = x5[528]._super._super;
  Val x504 = x5[529]._super._super;
  Val x505 = x5[530]._super._super;
  Val x506 = x5[531]._super._super;
  Val x507 = x5[532]._super._super;
  Val x508 = x5[533]._super._super;
  Val x509 = x5[534]._super._super;
  Val x510 = x5[535]._super._super;
  Val x511 = x5[536]._super._super;
  Val x512 = x5[537]._super._super;
  Val x513 = x5[538]._super._super;
  Val x514 = x5[539]._super._super;
  Val x515 = x5[540]._super._super;
  Val x516 = x5[541]._super._super;
  Val x517 = x5[542]._super._super;
  Val x518 = x5[543]._super._super;
  Val x519 = x6[736]._super._super;
  Val x520 = x6[737]._super._super;
  Val x521 = x6[738]._super._super;
  Val x522 = x6[739]._super._super;
  Val x523 = x6[740]._super._super;
  Val x524 = x6[741]._super._super;
  Val x525 = x6[742]._super._super;
  Val x526 = x6[743]._super._super;
  Val x527 = x6[744]._super._super;
  Val x528 = x6[745]._super._super;
  Val x529 = x6[746]._super._super;
  Val x530 = x6[747]._super._super;
  Val x531 = x6[748]._super._super;
  Val x532 = x6[749]._super._super;
  Val x533 = x6[750]._super._super;
  Val x534 = x6[751]._super._super;
  Val x535 = x6[752]._super._super;
  Val x536 = x6[753]._super._super;
  Val x537 = x6[754]._super._super;
  Val x538 = x6[755]._super._super;
  Val x539 = x6[756]._super._super;
  Val x540 = x6[757]._super._super;
  Val x541 = x6[758]._super._super;
  Val x542 = x6[759]._super._super;
  Val x543 = x6[760]._super._super;
  Val x544 = x6[761]._super._super;
  Val x545 = x6[762]._super._super;
  Val x546 = x6[763]._super._super;
  Val x547 = x6[764]._super._super;
  Val x548 = x6[765]._super._super;
  Val x549 = x6[766]._super._super;
  Val x550 = x6[767]._super._super;
  Val x551 = x6[704]._super._super;
  Val x552 = x6[705]._super._super;
  Val x553 = x6[706]._super._super;
  Val x554 = x6[707]._super._super;
  Val x555 = x6[708]._super._super;
  Val x556 = x6[709]._super._super;
  Val x557 = x6[710]._super._super;
  Val x558 = x6[711]._super._super;
  Val x559 = x6[712]._super._super;
  Val x560 = x6[713]._super._super;
  Val x561 = x6[714]._super._super;
  Val x562 = x6[715]._super._super;
  Val x563 = x6[716]._super._super;
  Val x564 = x6[717]._super._super;
  Val x565 = x6[718]._super._super;
  Val x566 = x6[719]._super._super;
  Val x567 = x6[720]._super._super;
  Val x568 = x6[721]._super._super;
  Val x569 = x6[722]._super._super;
  Val x570 = x6[723]._super._super;
  Val x571 = x6[724]._super._super;
  Val x572 = x6[725]._super._super;
  Val x573 = x6[726]._super._super;
  Val x574 = x6[727]._super._super;
  Val x575 = x6[728]._super._super;
  Val x576 = x6[729]._super._super;
  Val x577 = x6[730]._super._super;
  Val x578 = x6[731]._super._super;
  Val x579 = x6[732]._super._super;
  Val x580 = x6[733]._super._super;
  Val x581 = x6[734]._super._super;
  Val x582 = x6[735]._super._super;
  Val x583 = x6[672]._super._super;
  Val x584 = x6[673]._super._super;
  Val x585 = x6[674]._super._super;
  Val x586 = x6[675]._super._super;
  Val x587 = x6[676]._super._super;
  Val x588 = x6[677]._super._super;
  Val x589 = x6[678]._super._super;
  Val x590 = x6[679]._super._super;
  Val x591 = x6[680]._super._super;
  Val x592 = x6[681]._super._super;
  Val x593 = x6[682]._super._super;
  Val x594 = x6[683]._super._super;
  Val x595 = x6[684]._super._super;
  Val x596 = x6[685]._super._super;
  Val x597 = x6[686]._super._super;
  Val x598 = x6[687]._super._super;
  Val x599 = x6[688]._super._super;
  Val x600 = x6[689]._super._super;
  Val x601 = x6[690]._super._super;
  Val x602 = x6[691]._super._super;
  Val x603 = x6[692]._super._super;
  Val x604 = x6[693]._super._super;
  Val x605 = x6[694]._super._super;
  Val x606 = x6[695]._super._super;
  Val x607 = x6[696]._super._super;
  Val x608 = x6[697]._super._super;
  Val x609 = x6[698]._super._super;
  Val x610 = x6[699]._super._super;
  Val x611 = x6[700]._super._super;
  Val x612 = x6[701]._super._super;
  Val x613 = x6[702]._super._super;
  Val x614 = x6[703]._super._super;
  Val x615 = x6[640]._super._super;
  Val x616 = x6[641]._super._super;
  Val x617 = x6[642]._super._super;
  Val x618 = x6[643]._super._super;
  Val x619 = x6[644]._super._super;
  Val x620 = x6[645]._super._super;
  Val x621 = x6[646]._super._super;
  Val x622 = x6[647]._super._super;
  Val x623 = x6[648]._super._super;
  Val x624 = x6[649]._super._super;
  Val x625 = x6[650]._super._super;
  Val x626 = x6[651]._super._super;
  Val x627 = x6[652]._super._super;
  Val x628 = x6[653]._super._super;
  Val x629 = x6[654]._super._super;
  Val x630 = x6[655]._super._super;
  Val x631 = x6[656]._super._super;
  Val x632 = x6[657]._super._super;
  Val x633 = x6[658]._super._super;
  Val x634 = x6[659]._super._super;
  Val x635 = x6[660]._super._super;
  Val x636 = x6[661]._super._super;
  Val x637 = x6[662]._super._super;
  Val x638 = x6[663]._super._super;
  Val x639 = x6[664]._super._super;
  Val x640 = x6[665]._super._super;
  Val x641 = x6[666]._super._super;
  Val x642 = x6[667]._super._super;
  Val x643 = x6[668]._super._super;
  Val x644 = x6[669]._super._super;
  Val x645 = x6[670]._super._super;
  Val x646 = x6[671]._super._super;
  Val x647 = x6[608]._super._super;
  Val x648 = x6[609]._super._super;
  Val x649 = x6[610]._super._super;
  Val x650 = x6[611]._super._super;
  Val x651 = x6[612]._super._super;
  Val x652 = x6[613]._super._super;
  Val x653 = x6[614]._super._super;
  Val x654 = x6[615]._super._super;
  Val x655 = x6[616]._super._super;
  Val x656 = x6[617]._super._super;
  Val x657 = x6[618]._super._super;
  Val x658 = x6[619]._super._super;
  Val x659 = x6[620]._super._super;
  Val x660 = x6[621]._super._super;
  Val x661 = x6[622]._super._super;
  Val x662 = x6[623]._super._super;
  Val x663 = x6[624]._super._super;
  Val x664 = x6[625]._super._super;
  Val x665 = x6[626]._super._super;
  Val x666 = x6[627]._super._super;
  Val x667 = x6[628]._super._super;
  Val x668 = x6[629]._super._super;
  Val x669 = x6[630]._super._super;
  Val x670 = x6[631]._super._super;
  Val x671 = x6[632]._super._super;
  Val x672 = x6[633]._super._super;
  Val x673 = x6[634]._super._super;
  Val x674 = x6[635]._super._super;
  Val x675 = x6[636]._super._super;
  Val x676 = x6[637]._super._super;
  Val x677 = x6[638]._super._super;
  Val x678 = x6[639]._super._super;
  Val x679 = x6[576]._super._super;
  Val x680 = x6[577]._super._super;
  Val x681 = x6[578]._super._super;
  Val x682 = x6[579]._super._super;
  Val x683 = x6[580]._super._super;
  Val x684 = x6[581]._super._super;
  Val x685 = x6[582]._super._super;
  Val x686 = x6[583]._super._super;
  Val x687 = x6[584]._super._super;
  Val x688 = x6[585]._super._super;
  Val x689 = x6[586]._super._super;
  Val x690 = x6[587]._super._super;
  Val x691 = x6[588]._super._super;
  Val x692 = x6[589]._super._super;
  Val x693 = x6[590]._super._super;
  Val x694 = x6[591]._super._super;
  Val x695 = x6[592]._super._super;
  Val x696 = x6[593]._super._super;
  Val x697 = x6[594]._super._super;
  Val x698 = x6[595]._super._super;
  Val x699 = x6[596]._super._super;
  Val x700 = x6[597]._super._super;
  Val x701 = x6[598]._super._super;
  Val x702 = x6[599]._super._super;
  Val x703 = x6[600]._super._super;
  Val x704 = x6[601]._super._super;
  Val x705 = x6[602]._super._super;
  Val x706 = x6[603]._super._super;
  Val x707 = x6[604]._super._super;
  Val x708 = x6[605]._super._super;
  Val x709 = x6[606]._super._super;
  Val x710 = x6[607]._super._super;
  Val x711 = x6[544]._super._super;
  Val x712 = x6[545]._super._super;
  Val x713 = x6[546]._super._super;
  Val x714 = x6[547]._super._super;
  Val x715 = x6[548]._super._super;
  Val x716 = x6[549]._super._super;
  Val x717 = x6[550]._super._super;
  Val x718 = x6[551]._super._super;
  Val x719 = x6[552]._super._super;
  Val x720 = x6[553]._super._super;
  Val x721 = x6[554]._super._super;
  Val x722 = x6[555]._super._super;
  Val x723 = x6[556]._super._super;
  Val x724 = x6[557]._super._super;
  Val x725 = x6[558]._super._super;
  Val x726 = x6[559]._super._super;
  Val x727 = x6[560]._super._super;
  Val x728 = x6[561]._super._super;
  Val x729 = x6[562]._super._super;
  Val x730 = x6[563]._super._super;
  Val x731 = x6[564]._super._super;
  Val x732 = x6[565]._super._super;
  Val x733 = x6[566]._super._super;
  Val x734 = x6[567]._super._super;
  Val x735 = x6[568]._super._super;
  Val x736 = x6[569]._super._super;
  Val x737 = x6[570]._super._super;
  Val x738 = x6[571]._super._super;
  Val x739 = x6[572]._super._super;
  Val x740 = x6[573]._super._super;
  Val x741 = x6[574]._super._super;
  Val x742 = x6[575]._super._super;
  Val x743 = x6[512]._super._super;
  Val x744 = x6[513]._super._super;
  Val x745 = x6[514]._super._super;
  Val x746 = x6[515]._super._super;
  Val x747 = x6[516]._super._super;
  Val x748 = x6[517]._super._super;
  Val x749 = x6[518]._super._super;
  Val x750 = x6[519]._super._super;
  Val x751 = x6[520]._super._super;
  Val x752 = x6[521]._super._super;
  Val x753 = x6[522]._super._super;
  Val x754 = x6[523]._super._super;
  Val x755 = x6[524]._super._super;
  Val x756 = x6[525]._super._super;
  Val x757 = x6[526]._super._super;
  Val x758 = x6[527]._super._super;
  Val x759 = x6[528]._super._super;
  Val x760 = x6[529]._super._super;
  Val x761 = x6[530]._super._super;
  Val x762 = x6[531]._super._super;
  Val x763 = x6[532]._super._super;
  Val x764 = x6[533]._super._super;
  Val x765 = x6[534]._super._super;
  Val x766 = x6[535]._super._super;
  Val x767 = x6[536]._super._super;
  Val x768 = x6[537]._super._super;
  Val x769 = x6[538]._super._super;
  Val x770 = x6[539]._super._super;
  Val x771 = x6[540]._super._super;
  Val x772 = x6[541]._super._super;
  Val x773 = x6[542]._super._super;
  Val x774 = x6[543]._super._super;
  // ShaState(zirgen/circuit/keccak2/top.zir:130)
  // LoadShaState(zirgen/circuit/keccak2/top.zir:157)
  ShaStateStruct x775 = ShaStateStruct{
      .a = Val32Array4Array{Val32Array{x7,  x8,  x9,  x10, x11, x12, x13, x14, x15, x16, x17,
                                       x18, x19, x20, x21, x22, x23, x24, x25, x26, x27, x28,
                                       x29, x30, x31, x32, x33, x34, x35, x36, x37, x38},
                            Val32Array{x39, x40, x41, x42, x43, x44, x45, x46, x47, x48, x49,
                                       x50, x51, x52, x53, x54, x55, x56, x57, x58, x59, x60,
                                       x61, x62, x63, x64, x65, x66, x67, x68, x69, x70},
                            Val32Array{x71, x72, x73, x74, x75, x76, x77, x78,  x79,  x80, x81,
                                       x82, x83, x84, x85, x86, x87, x88, x89,  x90,  x91, x92,
                                       x93, x94, x95, x96, x97, x98, x99, x100, x101, x102},
                            Val32Array{x103, x104, x105, x106, x107, x108, x109, x110,
                                       x111, x112, x113, x114, x115, x116, x117, x118,
                                       x119, x120, x121, x122, x123, x124, x125, x126,
                                       x127, x128, x129, x130, x131, x132, x133, x134}},
      .e =
          Val32Array4Array{
              Val32Array{x135, x136, x137, x138, x139, x140, x141, x142, x143, x144, x145,
                         x146, x147, x148, x149, x150, x151, x152, x153, x154, x155, x156,
                         x157, x158, x159, x160, x161, x162, x163, x164, x165, x166},
              Val32Array{x167, x168, x169, x170, x171, x172, x173, x174, x175, x176, x177,
                         x178, x179, x180, x181, x182, x183, x184, x185, x186, x187, x188,
                         x189, x190, x191, x192, x193, x194, x195, x196, x197, x198},
              Val32Array{x199, x200, x201, x202, x203, x204, x205, x206, x207, x208, x209,
                         x210, x211, x212, x213, x214, x215, x216, x217, x218, x219, x220,
                         x221, x222, x223, x224, x225, x226, x227, x228, x229, x230},
              Val32Array{x231, x232, x233, x234, x235, x236, x237, x238, x239, x240, x241,
                         x242, x243, x244, x245, x246, x247, x248, x249, x250, x251, x252,
                         x253, x254, x255, x256, x257, x258, x259, x260, x261, x262}},
      .w = Val32Array16Array{
          Val32Array{x263, x264, x265, x266, x267, x268, x269, x270, x271, x272, x273,
                     x274, x275, x276, x277, x278, x279, x280, x281, x282, x283, x284,
                     x285, x286, x287, x288, x289, x290, x291, x292, x293, x294},
          Val32Array{x295, x296, x297, x298, x299, x300, x301, x302, x303, x304, x305,
                     x306, x307, x308, x309, x310, x311, x312, x313, x314, x315, x316,
                     x317, x318, x319, x320, x321, x322, x323, x324, x325, x326},
          Val32Array{x327, x328, x329, x330, x331, x332, x333, x334, x335, x336, x337,
                     x338, x339, x340, x341, x342, x343, x344, x345, x346, x347, x348,
                     x349, x350, x351, x352, x353, x354, x355, x356, x357, x358},
          Val32Array{x359, x360, x361, x362, x363, x364, x365, x366, x367, x368, x369,
                     x370, x371, x372, x373, x374, x375, x376, x377, x378, x379, x380,
                     x381, x382, x383, x384, x385, x386, x387, x388, x389, x390},
          Val32Array{x391, x392, x393, x394, x395, x396, x397, x398, x399, x400, x401,
                     x402, x403, x404, x405, x406, x407, x408, x409, x410, x411, x412,
                     x413, x414, x415, x416, x417, x418, x419, x420, x421, x422},
          Val32Array{x423, x424, x425, x426, x427, x428, x429, x430, x431, x432, x433,
                     x434, x435, x436, x437, x438, x439, x440, x441, x442, x443, x444,
                     x445, x446, x447, x448, x449, x450, x451, x452, x453, x454},
          Val32Array{x455, x456, x457, x458, x459, x460, x461, x462, x463, x464, x465,
                     x466, x467, x468, x469, x470, x471, x472, x473, x474, x475, x476,
                     x477, x478, x479, x480, x481, x482, x483, x484, x485, x486},
          Val32Array{x487, x488, x489, x490, x491, x492, x493, x494, x495, x496, x497,
                     x498, x499, x500, x501, x502, x503, x504, x505, x506, x507, x508,
                     x509, x510, x511, x512, x513, x514, x515, x516, x517, x518},
          Val32Array{x519, x520, x521, x522, x523, x524, x525, x526, x527, x528, x529,
                     x530, x531, x532, x533, x534, x535, x536, x537, x538, x539, x540,
                     x541, x542, x543, x544, x545, x546, x547, x548, x549, x550},
          Val32Array{x551, x552, x553, x554, x555, x556, x557, x558, x559, x560, x561,
                     x562, x563, x564, x565, x566, x567, x568, x569, x570, x571, x572,
                     x573, x574, x575, x576, x577, x578, x579, x580, x581, x582},
          Val32Array{x583, x584, x585, x586, x587, x588, x589, x590, x591, x592, x593,
                     x594, x595, x596, x597, x598, x599, x600, x601, x602, x603, x604,
                     x605, x606, x607, x608, x609, x610, x611, x612, x613, x614},
          Val32Array{x615, x616, x617, x618, x619, x620, x621, x622, x623, x624, x625,
                     x626, x627, x628, x629, x630, x631, x632, x633, x634, x635, x636,
                     x637, x638, x639, x640, x641, x642, x643, x644, x645, x646},
          Val32Array{x647, x648, x649, x650, x651, x652, x653, x654, x655, x656, x657,
                     x658, x659, x660, x661, x662, x663, x664, x665, x666, x667, x668,
                     x669, x670, x671, x672, x673, x674, x675, x676, x677, x678},
          Val32Array{x679, x680, x681, x682, x683, x684, x685, x686, x687, x688, x689,
                     x690, x691, x692, x693, x694, x695, x696, x697, x698, x699, x700,
                     x701, x702, x703, x704, x705, x706, x707, x708, x709, x710},
          Val32Array{x711, x712, x713, x714, x715, x716, x717, x718, x719, x720, x721,
                     x722, x723, x724, x725, x726, x727, x728, x729, x730, x731, x732,
                     x733, x734, x735, x736, x737, x738, x739, x740, x741, x742},
          Val32Array{x743, x744, x745, x746, x747, x748, x749, x750, x751, x752, x753,
                     x754, x755, x756, x757, x758, x759, x760, x761, x762, x763, x764,
                     x765, x766, x767, x768, x769, x770, x771, x772, x773, x774}}};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:190)
  OneHot_8_Struct x776 = exec_OneHot_8_(ctx, arg2, LAYOUT_LOOKUP(layout4, oneHot));
  NondetRegStruct8Array x777 = x776._super;
  Val x778 = x777[0]._super;
  Val x779 = x777[1]._super;
  Val x780 = x777[2]._super;
  Val x781 = x777[3]._super;
  Val x782 = x777[4]._super;
  Val x783 = x777[5]._super;
  Val x784 = x777[6]._super;
  Val x785 = x777[7]._super;
  Val x786;
  if (to_size_t(x778)) {
    x786 = Val(1);
  } else if (to_size_t(x779)) {
    x786 = Val(1);
  } else if (to_size_t(x780)) {
    x786 = Val(0);
  } else if (to_size_t(x781)) {
    x786 = Val(0);
  } else if (to_size_t(x782)) {
    x786 = Val(0);
  } else if (to_size_t(x783)) {
    x786 = Val(0);
  } else if (to_size_t(x784)) {
    x786 = Val(0);
  } else if (to_size_t(x785)) {
    x786 = Val(0);
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  // ShaCycle(zirgen/circuit/keccak2/top.zir:191)
  NondetRegStruct x787 = exec_Reg(ctx, x786, LAYOUT_LOOKUP(layout4, isLoad));
  // GetK8(zirgen/circuit/keccak2/top.zir:125)
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val x788 = (((x778 * Val(12184)) + (x779 * Val(43672))) + (x780 * Val(27073)));
  Val x789 = (((x788 + (x781 * Val(20818))) + (x782 * Val(2693))) + (x783 * Val(59553)));
  Val x790 = (((x778 * Val(17034)) + (x779 * Val(55303))) + (x780 * Val(58523)));
  Val x791 = (((x790 + (x781 * Val(38974))) + (x782 * Val(10167))) + (x783 * Val(41663)));
  Val x792 = (((x778 * Val(17553)) + (x779 * Val(23297))) + (x780 * Val(18310)));
  Val x793 = (((x792 + (x781 * Val(50797))) + (x782 * Val(8504))) + (x783 * Val(26187)));
  Val x794 = (((x778 * Val(28983)) + (x779 * Val(4739))) + (x780 * Val(61374)));
  Val x795 = (((x794 + (x781 * Val(43057))) + (x782 * Val(11803))) + (x783 * Val(43034)));
  Val x796 = (((x778 * Val(64463)) + (x779 * Val(34238))) + (x780 * Val(40390)));
  Val x797 = (((x796 + (x781 * Val(10184))) + (x782 * Val(28156))) + (x783 * Val(35696)));
  Val x798 = (((x778 * Val(46528)) + (x779 * Val(9265))) + (x780 * Val(4033)));
  Val x799 = (((x798 + (x781 * Val(45059))) + (x782 * Val(19756))) + (x783 * Val(49739)));
  Val x800 = (((x778 * Val(56229)) + (x779 * Val(32195))) + (x780 * Val(41420)));
  Val x801 = (((x800 + (x781 * Val(32711))) + (x782 * Val(3347))) + (x783 * Val(20899)));
  Val x802 = (((x778 * Val(59829)) + (x779 * Val(21772))) + (x780 * Val(9228)));
  Val x803 = (((x802 + (x781 * Val(48985))) + (x782 * Val(21304))) + (x783 * Val(51052)));
  Val x804 = (((x778 * Val(49755)) + (x779 * Val(23924))) + (x780 * Val(11375)));
  Val x805 = (((x804 + (x781 * Val(3059))) + (x782 * Val(29524))) + (x783 * Val(59417)));
  Val x806 = (((x778 * Val(14678)) + (x779 * Val(29374))) + (x780 * Val(11753)));
  Val x807 = (((x806 + (x781 * Val(50912))) + (x782 * Val(25866))) + (x783 * Val(53650)));
  Val x808 = (((x778 * Val(4593)) + (x779 * Val(45566))) + (x780 * Val(33962)));
  Val x809 = (((x808 + (x781 * Val(37191))) + (x782 * Val(2747))) + (x783 * Val(1572)));
  Val x810 = (((x778 * Val(23025)) + (x779 * Val(32990))) + (x780 * Val(19060)));
  Val x811 = (((x810 + (x781 * Val(54695))) + (x782 * Val(30314))) + (x783 * Val(54937)));
  Val x812 = (((x778 * Val(33444)) + (x779 * Val(1703))) + (x780 * Val(43484)));
  Val x813 = (((x812 + (x781 * Val(25425))) + (x782 * Val(51502))) + (x783 * Val(13701)));
  Val x814 = (((x778 * Val(37439)) + (x779 * Val(39900))) + (x780 * Val(23728)));
  Val x815 = (((x814 + (x781 * Val(1738))) + (x782 * Val(33218))) + (x783 * Val(62478)));
  Val x816 = (((x778 * Val(24277)) + (x779 * Val(61812))) + (x780 * Val(35034)));
  Val x817 = (((x816 + (x781 * Val(10599))) + (x782 * Val(11397))) + (x783 * Val(41072)));
  Val x818 = (((x778 * Val(43804)) + (x779 * Val(49563))) + (x780 * Val(30457)));
  Val x819 = (((x818 + (x781 * Val(5161))) + (x782 * Val(37490))) + (x783 * Val(4202)));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:186)
  TopStateKflatStruct100Array x820 = arg0.kflat;
  Val x821 = x820[0]._super._super;
  Val x822 = x820[1]._super._super;
  Val x823 = x820[2]._super._super;
  Val x824 = x820[3]._super._super;
  Val x825 = x820[4]._super._super;
  Val x826 = x820[5]._super._super;
  Val x827 = x820[6]._super._super;
  Val x828 = x820[7]._super._super;
  Val x829 = x820[8]._super._super;
  Val x830 = x820[9]._super._super;
  Val x831 = x820[10]._super._super;
  Val x832 = x820[11]._super._super;
  Val x833 = x820[12]._super._super;
  Val x834 = x820[13]._super._super;
  Val x835 = x820[14]._super._super;
  Val x836 = x820[15]._super._super;
  Val x837 = x820[16]._super._super;
  Val x838 = x820[17]._super._super;
  Val x839 = x820[18]._super._super;
  Val x840 = x820[19]._super._super;
  Val x841 = x820[20]._super._super;
  Val x842 = x820[21]._super._super;
  Val x843 = x820[22]._super._super;
  Val x844 = x820[23]._super._super;
  Val x845 = x820[24]._super._super;
  Val x846 = x820[25]._super._super;
  Val x847 = x820[26]._super._super;
  Val x848 = x820[27]._super._super;
  Val x849 = x820[28]._super._super;
  Val x850 = x820[29]._super._super;
  Val x851 = x820[30]._super._super;
  Val x852 = x820[31]._super._super;
  Val x853 = x820[32]._super._super;
  Val x854 = x820[33]._super._super;
  Val x855 = x820[34]._super._super;
  Val x856 = x820[35]._super._super;
  Val x857 = x820[36]._super._super;
  Val x858 = x820[37]._super._super;
  Val x859 = x820[38]._super._super;
  Val x860 = x820[39]._super._super;
  Val x861 = x820[40]._super._super;
  Val x862 = x820[41]._super._super;
  Val x863 = x820[42]._super._super;
  Val x864 = x820[43]._super._super;
  Val x865 = x820[44]._super._super;
  Val x866 = x820[45]._super._super;
  Val x867 = x820[46]._super._super;
  Val x868 = x820[47]._super._super;
  Val x869 = x820[48]._super._super;
  Val x870 = x820[49]._super._super;
  Val x871 = x820[50]._super._super;
  Val x872 = x820[51]._super._super;
  Val x873 = x820[52]._super._super;
  Val x874 = x820[53]._super._super;
  Val x875 = x820[54]._super._super;
  Val x876 = x820[55]._super._super;
  Val x877 = x820[56]._super._super;
  Val x878 = x820[57]._super._super;
  Val x879 = x820[58]._super._super;
  Val x880 = x820[59]._super._super;
  Val x881 = x820[60]._super._super;
  Val x882 = x820[61]._super._super;
  Val x883 = x820[62]._super._super;
  Val x884 = x820[63]._super._super;
  Val x885 = x820[64]._super._super;
  Val x886 = x820[65]._super._super;
  Val x887 = x820[66]._super._super;
  Val x888 = x820[67]._super._super;
  Val x889 = x820[68]._super._super;
  Val x890 = x820[69]._super._super;
  Val x891 = x820[70]._super._super;
  Val x892 = x820[71]._super._super;
  Val x893 = x820[72]._super._super;
  Val x894 = x820[73]._super._super;
  Val x895 = x820[74]._super._super;
  Val x896 = x820[75]._super._super;
  Val x897 = x820[76]._super._super;
  Val x898 = x820[77]._super._super;
  Val x899 = x820[78]._super._super;
  Val x900 = x820[79]._super._super;
  Val x901 = x820[80]._super._super;
  Val x902 = x820[81]._super._super;
  Val x903 = x820[82]._super._super;
  Val x904 = x820[83]._super._super;
  Val x905 = x820[84]._super._super;
  Val x906 = x820[85]._super._super;
  Val x907 = x820[86]._super._super;
  Val x908 = x820[87]._super._super;
  Val x909 = x820[88]._super._super;
  Val x910 = x820[89]._super._super;
  Val x911 = x820[90]._super._super;
  Val x912 = x820[91]._super._super;
  Val x913 = x820[92]._super._super;
  Val x914 = x820[93]._super._super;
  Val x915 = x820[94]._super._super;
  Val x916 = x820[95]._super._super;
  Val x917 = x820[96]._super._super;
  Val x918 = x820[97]._super._super;
  Val x919 = x820[98]._super._super;
  Val x920 = x820[99]._super._super;
  Val100Array x921 = Val100Array{
      x821, x822, x823, x824, x825, x826, x827, x828, x829, x830, x831, x832, x833, x834, x835,
      x836, x837, x838, x839, x840, x841, x842, x843, x844, x845, x846, x847, x848, x849, x850,
      x851, x852, x853, x854, x855, x856, x857, x858, x859, x860, x861, x862, x863, x864, x865,
      x866, x867, x868, x869, x870, x871, x872, x873, x874, x875, x876, x877, x878, x879, x880,
      x881, x882, x883, x884, x885, x886, x887, x888, x889, x890, x891, x892, x893, x894, x895,
      x896, x897, x898, x899, x900, x901, x902, x903, x904, x905, x906, x907, x908, x909, x910,
      x911, x912, x913, x914, x915, x916, x917, x918, x919, x920};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:193)
  LoadWinStruct x922 = exec_LoadWin(ctx, x921, x776, arg3, LAYOUT_LOOKUP(layout4, win));
  Val2Array8Array x923 = x922._super;
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x924 = Val2Array{((x789 + (x784 * Val(49430))) + (x785 * Val(33518))),
                             ((x791 + (x784 * Val(6564))) + (x785 * Val(29839)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:191)
  Val x925 = x787._super;
  // ShaCycle(zirgen/circuit/keccak2/top.zir:194)
  DoShaStepStruct x926 =
      exec_DoShaStep(ctx, x775, x924, x925, x923[0], LAYOUT_LOOKUP(layout4, step0));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x927 = Val2Array{((x793 + (x784 * Val(27656))) + (x785 * Val(25455))),
                             ((x795 + (x784 * Val(7735))) + (x785 * Val(30885)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:195)
  DoShaStepStruct x928 =
      exec_DoShaStep(ctx, x926.newState, x927, x925, x923[1], LAYOUT_LOOKUP(layout4, step1));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x929 = Val2Array{((x797 + (x784 * Val(30540))) + (x785 * Val(30740))),
                             ((x799 + (x784 * Val(10056))) + (x785 * Val(33992)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:196)
  DoShaStepStruct x930 =
      exec_DoShaStep(ctx, x928.newState, x929, x925, x923[2], LAYOUT_LOOKUP(layout4, step2));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x931 = Val2Array{((x801 + (x784 * Val(48309))) + (x785 * Val(520))),
                             ((x803 + (x784 * Val(13488))) + (x785 * Val(36039)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:197)
  DoShaStepStruct x932 =
      exec_DoShaStep(ctx, x930.newState, x931, x925, x923[3], LAYOUT_LOOKUP(layout4, step3));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x933 = Val2Array{((x805 + (x784 * Val(3251))) + (x785 * Val(65530))),
                             ((x807 + (x784 * Val(14620))) + (x785 * Val(37054)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:198)
  DoShaStepStruct x934 =
      exec_DoShaStep(ctx, x932.newState, x933, x925, x923[4], LAYOUT_LOOKUP(layout4, step4));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x935 = Val2Array{((x809 + (x784 * Val(43594))) + (x785 * Val(27883))),
                             ((x811 + (x784 * Val(20184))) + (x785 * Val(42064)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:199)
  DoShaStepStruct x936 =
      exec_DoShaStep(ctx, x934.newState, x935, x925, x923[5], LAYOUT_LOOKUP(layout4, step5));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x937 = Val2Array{((x813 + (x784 * Val(51791))) + (x785 * Val(41975))),
                             ((x815 + (x784 * Val(23452))) + (x785 * Val(48889)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:200)
  DoShaStepStruct x938 =
      exec_DoShaStep(ctx, x936.newState, x937, x925, x923[6], LAYOUT_LOOKUP(layout4, step6));
  // ShaCycle(zirgen/circuit/keccak2/top.zir:192)
  Val2Array x939 = Val2Array{((x817 + (x784 * Val(28659))) + (x785 * Val(30962))),
                             ((x819 + (x784 * Val(26670))) + (x785 * Val(50801)))};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:201)
  DoShaStepStruct x940 =
      exec_DoShaStep(ctx, x938.newState, x939, x925, x923[7], LAYOUT_LOOKUP(layout4, step7));
  // ExtractBits(zirgen/circuit/keccak2/top.zir:161)
  // ShaCycle(zirgen/circuit/keccak2/top.zir:206)
  UnpackReg_32__16__SuperStruct32Array x941 = x926.a._super;
  Val x942 = x941[0]._super._super;
  Val x943 = x941[1]._super._super;
  Val x944 = x941[2]._super._super;
  Val x945 = x941[3]._super._super;
  Val x946 = x941[4]._super._super;
  Val x947 = x941[5]._super._super;
  Val x948 = x941[6]._super._super;
  Val x949 = x941[7]._super._super;
  Val x950 = x941[8]._super._super;
  Val x951 = x941[9]._super._super;
  Val x952 = x941[10]._super._super;
  Val x953 = x941[11]._super._super;
  Val x954 = x941[12]._super._super;
  Val x955 = x941[13]._super._super;
  Val x956 = x941[14]._super._super;
  Val x957 = x941[15]._super._super;
  Val x958 = x941[16]._super._super;
  Val x959 = x941[17]._super._super;
  Val x960 = x941[18]._super._super;
  Val x961 = x941[19]._super._super;
  Val x962 = x941[20]._super._super;
  Val x963 = x941[21]._super._super;
  Val x964 = x941[22]._super._super;
  Val x965 = x941[23]._super._super;
  Val x966 = x941[24]._super._super;
  Val x967 = x941[25]._super._super;
  Val x968 = x941[26]._super._super;
  Val x969 = x941[27]._super._super;
  Val x970 = x941[28]._super._super;
  Val x971 = x941[29]._super._super;
  Val x972 = x941[30]._super._super;
  Val x973 = x941[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x974 = x928.a._super;
  Val x975 = x974[0]._super._super;
  Val x976 = x974[1]._super._super;
  Val x977 = x974[2]._super._super;
  Val x978 = x974[3]._super._super;
  Val x979 = x974[4]._super._super;
  Val x980 = x974[5]._super._super;
  Val x981 = x974[6]._super._super;
  Val x982 = x974[7]._super._super;
  Val x983 = x974[8]._super._super;
  Val x984 = x974[9]._super._super;
  Val x985 = x974[10]._super._super;
  Val x986 = x974[11]._super._super;
  Val x987 = x974[12]._super._super;
  Val x988 = x974[13]._super._super;
  Val x989 = x974[14]._super._super;
  Val x990 = x974[15]._super._super;
  Val x991 = x974[16]._super._super;
  Val x992 = x974[17]._super._super;
  Val x993 = x974[18]._super._super;
  Val x994 = x974[19]._super._super;
  Val x995 = x974[20]._super._super;
  Val x996 = x974[21]._super._super;
  Val x997 = x974[22]._super._super;
  Val x998 = x974[23]._super._super;
  Val x999 = x974[24]._super._super;
  Val x1000 = x974[25]._super._super;
  Val x1001 = x974[26]._super._super;
  Val x1002 = x974[27]._super._super;
  Val x1003 = x974[28]._super._super;
  Val x1004 = x974[29]._super._super;
  Val x1005 = x974[30]._super._super;
  Val x1006 = x974[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1007 = x930.a._super;
  Val x1008 = x1007[0]._super._super;
  Val x1009 = x1007[1]._super._super;
  Val x1010 = x1007[2]._super._super;
  Val x1011 = x1007[3]._super._super;
  Val x1012 = x1007[4]._super._super;
  Val x1013 = x1007[5]._super._super;
  Val x1014 = x1007[6]._super._super;
  Val x1015 = x1007[7]._super._super;
  Val x1016 = x1007[8]._super._super;
  Val x1017 = x1007[9]._super._super;
  Val x1018 = x1007[10]._super._super;
  Val x1019 = x1007[11]._super._super;
  Val x1020 = x1007[12]._super._super;
  Val x1021 = x1007[13]._super._super;
  Val x1022 = x1007[14]._super._super;
  Val x1023 = x1007[15]._super._super;
  Val x1024 = x1007[16]._super._super;
  Val x1025 = x1007[17]._super._super;
  Val x1026 = x1007[18]._super._super;
  Val x1027 = x1007[19]._super._super;
  Val x1028 = x1007[20]._super._super;
  Val x1029 = x1007[21]._super._super;
  Val x1030 = x1007[22]._super._super;
  Val x1031 = x1007[23]._super._super;
  Val x1032 = x1007[24]._super._super;
  Val x1033 = x1007[25]._super._super;
  Val x1034 = x1007[26]._super._super;
  Val x1035 = x1007[27]._super._super;
  Val x1036 = x1007[28]._super._super;
  Val x1037 = x1007[29]._super._super;
  Val x1038 = x1007[30]._super._super;
  Val x1039 = x1007[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1040 = x932.a._super;
  Val x1041 = x1040[0]._super._super;
  Val x1042 = x1040[1]._super._super;
  Val x1043 = x1040[2]._super._super;
  Val x1044 = x1040[3]._super._super;
  Val x1045 = x1040[4]._super._super;
  Val x1046 = x1040[5]._super._super;
  Val x1047 = x1040[6]._super._super;
  Val x1048 = x1040[7]._super._super;
  Val x1049 = x1040[8]._super._super;
  Val x1050 = x1040[9]._super._super;
  Val x1051 = x1040[10]._super._super;
  Val x1052 = x1040[11]._super._super;
  Val x1053 = x1040[12]._super._super;
  Val x1054 = x1040[13]._super._super;
  Val x1055 = x1040[14]._super._super;
  Val x1056 = x1040[15]._super._super;
  Val x1057 = x1040[16]._super._super;
  Val x1058 = x1040[17]._super._super;
  Val x1059 = x1040[18]._super._super;
  Val x1060 = x1040[19]._super._super;
  Val x1061 = x1040[20]._super._super;
  Val x1062 = x1040[21]._super._super;
  Val x1063 = x1040[22]._super._super;
  Val x1064 = x1040[23]._super._super;
  Val x1065 = x1040[24]._super._super;
  Val x1066 = x1040[25]._super._super;
  Val x1067 = x1040[26]._super._super;
  Val x1068 = x1040[27]._super._super;
  Val x1069 = x1040[28]._super._super;
  Val x1070 = x1040[29]._super._super;
  Val x1071 = x1040[30]._super._super;
  Val x1072 = x1040[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1073 = x934.a._super;
  Val x1074 = x1073[0]._super._super;
  Val x1075 = x1073[1]._super._super;
  Val x1076 = x1073[2]._super._super;
  Val x1077 = x1073[3]._super._super;
  Val x1078 = x1073[4]._super._super;
  Val x1079 = x1073[5]._super._super;
  Val x1080 = x1073[6]._super._super;
  Val x1081 = x1073[7]._super._super;
  Val x1082 = x1073[8]._super._super;
  Val x1083 = x1073[9]._super._super;
  Val x1084 = x1073[10]._super._super;
  Val x1085 = x1073[11]._super._super;
  Val x1086 = x1073[12]._super._super;
  Val x1087 = x1073[13]._super._super;
  Val x1088 = x1073[14]._super._super;
  Val x1089 = x1073[15]._super._super;
  Val x1090 = x1073[16]._super._super;
  Val x1091 = x1073[17]._super._super;
  Val x1092 = x1073[18]._super._super;
  Val x1093 = x1073[19]._super._super;
  Val x1094 = x1073[20]._super._super;
  Val x1095 = x1073[21]._super._super;
  Val x1096 = x1073[22]._super._super;
  Val x1097 = x1073[23]._super._super;
  Val x1098 = x1073[24]._super._super;
  Val x1099 = x1073[25]._super._super;
  Val x1100 = x1073[26]._super._super;
  Val x1101 = x1073[27]._super._super;
  Val x1102 = x1073[28]._super._super;
  Val x1103 = x1073[29]._super._super;
  Val x1104 = x1073[30]._super._super;
  Val x1105 = x1073[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1106 = x936.a._super;
  Val x1107 = x1106[0]._super._super;
  Val x1108 = x1106[1]._super._super;
  Val x1109 = x1106[2]._super._super;
  Val x1110 = x1106[3]._super._super;
  Val x1111 = x1106[4]._super._super;
  Val x1112 = x1106[5]._super._super;
  Val x1113 = x1106[6]._super._super;
  Val x1114 = x1106[7]._super._super;
  Val x1115 = x1106[8]._super._super;
  Val x1116 = x1106[9]._super._super;
  Val x1117 = x1106[10]._super._super;
  Val x1118 = x1106[11]._super._super;
  Val x1119 = x1106[12]._super._super;
  Val x1120 = x1106[13]._super._super;
  Val x1121 = x1106[14]._super._super;
  Val x1122 = x1106[15]._super._super;
  Val x1123 = x1106[16]._super._super;
  Val x1124 = x1106[17]._super._super;
  Val x1125 = x1106[18]._super._super;
  Val x1126 = x1106[19]._super._super;
  Val x1127 = x1106[20]._super._super;
  Val x1128 = x1106[21]._super._super;
  Val x1129 = x1106[22]._super._super;
  Val x1130 = x1106[23]._super._super;
  Val x1131 = x1106[24]._super._super;
  Val x1132 = x1106[25]._super._super;
  Val x1133 = x1106[26]._super._super;
  Val x1134 = x1106[27]._super._super;
  Val x1135 = x1106[28]._super._super;
  Val x1136 = x1106[29]._super._super;
  Val x1137 = x1106[30]._super._super;
  Val x1138 = x1106[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1139 = x938.a._super;
  Val x1140 = x1139[0]._super._super;
  Val x1141 = x1139[1]._super._super;
  Val x1142 = x1139[2]._super._super;
  Val x1143 = x1139[3]._super._super;
  Val x1144 = x1139[4]._super._super;
  Val x1145 = x1139[5]._super._super;
  Val x1146 = x1139[6]._super._super;
  Val x1147 = x1139[7]._super._super;
  Val x1148 = x1139[8]._super._super;
  Val x1149 = x1139[9]._super._super;
  Val x1150 = x1139[10]._super._super;
  Val x1151 = x1139[11]._super._super;
  Val x1152 = x1139[12]._super._super;
  Val x1153 = x1139[13]._super._super;
  Val x1154 = x1139[14]._super._super;
  Val x1155 = x1139[15]._super._super;
  Val x1156 = x1139[16]._super._super;
  Val x1157 = x1139[17]._super._super;
  Val x1158 = x1139[18]._super._super;
  Val x1159 = x1139[19]._super._super;
  Val x1160 = x1139[20]._super._super;
  Val x1161 = x1139[21]._super._super;
  Val x1162 = x1139[22]._super._super;
  Val x1163 = x1139[23]._super._super;
  Val x1164 = x1139[24]._super._super;
  Val x1165 = x1139[25]._super._super;
  Val x1166 = x1139[26]._super._super;
  Val x1167 = x1139[27]._super._super;
  Val x1168 = x1139[28]._super._super;
  Val x1169 = x1139[29]._super._super;
  Val x1170 = x1139[30]._super._super;
  Val x1171 = x1139[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1172 = x940.a._super;
  Val x1173 = x1172[0]._super._super;
  Val x1174 = x1172[1]._super._super;
  Val x1175 = x1172[2]._super._super;
  Val x1176 = x1172[3]._super._super;
  Val x1177 = x1172[4]._super._super;
  Val x1178 = x1172[5]._super._super;
  Val x1179 = x1172[6]._super._super;
  Val x1180 = x1172[7]._super._super;
  Val x1181 = x1172[8]._super._super;
  Val x1182 = x1172[9]._super._super;
  Val x1183 = x1172[10]._super._super;
  Val x1184 = x1172[11]._super._super;
  Val x1185 = x1172[12]._super._super;
  Val x1186 = x1172[13]._super._super;
  Val x1187 = x1172[14]._super._super;
  Val x1188 = x1172[15]._super._super;
  Val x1189 = x1172[16]._super._super;
  Val x1190 = x1172[17]._super._super;
  Val x1191 = x1172[18]._super._super;
  Val x1192 = x1172[19]._super._super;
  Val x1193 = x1172[20]._super._super;
  Val x1194 = x1172[21]._super._super;
  Val x1195 = x1172[22]._super._super;
  Val x1196 = x1172[23]._super._super;
  Val x1197 = x1172[24]._super._super;
  Val x1198 = x1172[25]._super._super;
  Val x1199 = x1172[26]._super._super;
  Val x1200 = x1172[27]._super._super;
  Val x1201 = x1172[28]._super._super;
  Val x1202 = x1172[29]._super._super;
  Val x1203 = x1172[30]._super._super;
  Val x1204 = x1172[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1205 = x926.e._super;
  Val x1206 = x1205[0]._super._super;
  Val x1207 = x1205[1]._super._super;
  Val x1208 = x1205[2]._super._super;
  Val x1209 = x1205[3]._super._super;
  Val x1210 = x1205[4]._super._super;
  Val x1211 = x1205[5]._super._super;
  Val x1212 = x1205[6]._super._super;
  Val x1213 = x1205[7]._super._super;
  Val x1214 = x1205[8]._super._super;
  Val x1215 = x1205[9]._super._super;
  Val x1216 = x1205[10]._super._super;
  Val x1217 = x1205[11]._super._super;
  Val x1218 = x1205[12]._super._super;
  Val x1219 = x1205[13]._super._super;
  Val x1220 = x1205[14]._super._super;
  Val x1221 = x1205[15]._super._super;
  Val x1222 = x1205[16]._super._super;
  Val x1223 = x1205[17]._super._super;
  Val x1224 = x1205[18]._super._super;
  Val x1225 = x1205[19]._super._super;
  Val x1226 = x1205[20]._super._super;
  Val x1227 = x1205[21]._super._super;
  Val x1228 = x1205[22]._super._super;
  Val x1229 = x1205[23]._super._super;
  Val x1230 = x1205[24]._super._super;
  Val x1231 = x1205[25]._super._super;
  Val x1232 = x1205[26]._super._super;
  Val x1233 = x1205[27]._super._super;
  Val x1234 = x1205[28]._super._super;
  Val x1235 = x1205[29]._super._super;
  Val x1236 = x1205[30]._super._super;
  Val x1237 = x1205[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1238 = x928.e._super;
  Val x1239 = x1238[0]._super._super;
  Val x1240 = x1238[1]._super._super;
  Val x1241 = x1238[2]._super._super;
  Val x1242 = x1238[3]._super._super;
  Val x1243 = x1238[4]._super._super;
  Val x1244 = x1238[5]._super._super;
  Val x1245 = x1238[6]._super._super;
  Val x1246 = x1238[7]._super._super;
  Val x1247 = x1238[8]._super._super;
  Val x1248 = x1238[9]._super._super;
  Val x1249 = x1238[10]._super._super;
  Val x1250 = x1238[11]._super._super;
  Val x1251 = x1238[12]._super._super;
  Val x1252 = x1238[13]._super._super;
  Val x1253 = x1238[14]._super._super;
  Val x1254 = x1238[15]._super._super;
  Val x1255 = x1238[16]._super._super;
  Val x1256 = x1238[17]._super._super;
  Val x1257 = x1238[18]._super._super;
  Val x1258 = x1238[19]._super._super;
  Val x1259 = x1238[20]._super._super;
  Val x1260 = x1238[21]._super._super;
  Val x1261 = x1238[22]._super._super;
  Val x1262 = x1238[23]._super._super;
  Val x1263 = x1238[24]._super._super;
  Val x1264 = x1238[25]._super._super;
  Val x1265 = x1238[26]._super._super;
  Val x1266 = x1238[27]._super._super;
  Val x1267 = x1238[28]._super._super;
  Val x1268 = x1238[29]._super._super;
  Val x1269 = x1238[30]._super._super;
  Val x1270 = x1238[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1271 = x930.e._super;
  Val x1272 = x1271[0]._super._super;
  Val x1273 = x1271[1]._super._super;
  Val x1274 = x1271[2]._super._super;
  Val x1275 = x1271[3]._super._super;
  Val x1276 = x1271[4]._super._super;
  Val x1277 = x1271[5]._super._super;
  Val x1278 = x1271[6]._super._super;
  Val x1279 = x1271[7]._super._super;
  Val x1280 = x1271[8]._super._super;
  Val x1281 = x1271[9]._super._super;
  Val x1282 = x1271[10]._super._super;
  Val x1283 = x1271[11]._super._super;
  Val x1284 = x1271[12]._super._super;
  Val x1285 = x1271[13]._super._super;
  Val x1286 = x1271[14]._super._super;
  Val x1287 = x1271[15]._super._super;
  Val x1288 = x1271[16]._super._super;
  Val x1289 = x1271[17]._super._super;
  Val x1290 = x1271[18]._super._super;
  Val x1291 = x1271[19]._super._super;
  Val x1292 = x1271[20]._super._super;
  Val x1293 = x1271[21]._super._super;
  Val x1294 = x1271[22]._super._super;
  Val x1295 = x1271[23]._super._super;
  Val x1296 = x1271[24]._super._super;
  Val x1297 = x1271[25]._super._super;
  Val x1298 = x1271[26]._super._super;
  Val x1299 = x1271[27]._super._super;
  Val x1300 = x1271[28]._super._super;
  Val x1301 = x1271[29]._super._super;
  Val x1302 = x1271[30]._super._super;
  Val x1303 = x1271[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1304 = x932.e._super;
  Val x1305 = x1304[0]._super._super;
  Val x1306 = x1304[1]._super._super;
  Val x1307 = x1304[2]._super._super;
  Val x1308 = x1304[3]._super._super;
  Val x1309 = x1304[4]._super._super;
  Val x1310 = x1304[5]._super._super;
  Val x1311 = x1304[6]._super._super;
  Val x1312 = x1304[7]._super._super;
  Val x1313 = x1304[8]._super._super;
  Val x1314 = x1304[9]._super._super;
  Val x1315 = x1304[10]._super._super;
  Val x1316 = x1304[11]._super._super;
  Val x1317 = x1304[12]._super._super;
  Val x1318 = x1304[13]._super._super;
  Val x1319 = x1304[14]._super._super;
  Val x1320 = x1304[15]._super._super;
  Val x1321 = x1304[16]._super._super;
  Val x1322 = x1304[17]._super._super;
  Val x1323 = x1304[18]._super._super;
  Val x1324 = x1304[19]._super._super;
  Val x1325 = x1304[20]._super._super;
  Val x1326 = x1304[21]._super._super;
  Val x1327 = x1304[22]._super._super;
  Val x1328 = x1304[23]._super._super;
  Val x1329 = x1304[24]._super._super;
  Val x1330 = x1304[25]._super._super;
  Val x1331 = x1304[26]._super._super;
  Val x1332 = x1304[27]._super._super;
  Val x1333 = x1304[28]._super._super;
  Val x1334 = x1304[29]._super._super;
  Val x1335 = x1304[30]._super._super;
  Val x1336 = x1304[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1337 = x934.e._super;
  Val x1338 = x1337[0]._super._super;
  Val x1339 = x1337[1]._super._super;
  Val x1340 = x1337[2]._super._super;
  Val x1341 = x1337[3]._super._super;
  Val x1342 = x1337[4]._super._super;
  Val x1343 = x1337[5]._super._super;
  Val x1344 = x1337[6]._super._super;
  Val x1345 = x1337[7]._super._super;
  Val x1346 = x1337[8]._super._super;
  Val x1347 = x1337[9]._super._super;
  Val x1348 = x1337[10]._super._super;
  Val x1349 = x1337[11]._super._super;
  Val x1350 = x1337[12]._super._super;
  Val x1351 = x1337[13]._super._super;
  Val x1352 = x1337[14]._super._super;
  Val x1353 = x1337[15]._super._super;
  Val x1354 = x1337[16]._super._super;
  Val x1355 = x1337[17]._super._super;
  Val x1356 = x1337[18]._super._super;
  Val x1357 = x1337[19]._super._super;
  Val x1358 = x1337[20]._super._super;
  Val x1359 = x1337[21]._super._super;
  Val x1360 = x1337[22]._super._super;
  Val x1361 = x1337[23]._super._super;
  Val x1362 = x1337[24]._super._super;
  Val x1363 = x1337[25]._super._super;
  Val x1364 = x1337[26]._super._super;
  Val x1365 = x1337[27]._super._super;
  Val x1366 = x1337[28]._super._super;
  Val x1367 = x1337[29]._super._super;
  Val x1368 = x1337[30]._super._super;
  Val x1369 = x1337[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1370 = x936.e._super;
  Val x1371 = x1370[0]._super._super;
  Val x1372 = x1370[1]._super._super;
  Val x1373 = x1370[2]._super._super;
  Val x1374 = x1370[3]._super._super;
  Val x1375 = x1370[4]._super._super;
  Val x1376 = x1370[5]._super._super;
  Val x1377 = x1370[6]._super._super;
  Val x1378 = x1370[7]._super._super;
  Val x1379 = x1370[8]._super._super;
  Val x1380 = x1370[9]._super._super;
  Val x1381 = x1370[10]._super._super;
  Val x1382 = x1370[11]._super._super;
  Val x1383 = x1370[12]._super._super;
  Val x1384 = x1370[13]._super._super;
  Val x1385 = x1370[14]._super._super;
  Val x1386 = x1370[15]._super._super;
  Val x1387 = x1370[16]._super._super;
  Val x1388 = x1370[17]._super._super;
  Val x1389 = x1370[18]._super._super;
  Val x1390 = x1370[19]._super._super;
  Val x1391 = x1370[20]._super._super;
  Val x1392 = x1370[21]._super._super;
  Val x1393 = x1370[22]._super._super;
  Val x1394 = x1370[23]._super._super;
  Val x1395 = x1370[24]._super._super;
  Val x1396 = x1370[25]._super._super;
  Val x1397 = x1370[26]._super._super;
  Val x1398 = x1370[27]._super._super;
  Val x1399 = x1370[28]._super._super;
  Val x1400 = x1370[29]._super._super;
  Val x1401 = x1370[30]._super._super;
  Val x1402 = x1370[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1403 = x938.e._super;
  Val x1404 = x1403[0]._super._super;
  Val x1405 = x1403[1]._super._super;
  Val x1406 = x1403[2]._super._super;
  Val x1407 = x1403[3]._super._super;
  Val x1408 = x1403[4]._super._super;
  Val x1409 = x1403[5]._super._super;
  Val x1410 = x1403[6]._super._super;
  Val x1411 = x1403[7]._super._super;
  Val x1412 = x1403[8]._super._super;
  Val x1413 = x1403[9]._super._super;
  Val x1414 = x1403[10]._super._super;
  Val x1415 = x1403[11]._super._super;
  Val x1416 = x1403[12]._super._super;
  Val x1417 = x1403[13]._super._super;
  Val x1418 = x1403[14]._super._super;
  Val x1419 = x1403[15]._super._super;
  Val x1420 = x1403[16]._super._super;
  Val x1421 = x1403[17]._super._super;
  Val x1422 = x1403[18]._super._super;
  Val x1423 = x1403[19]._super._super;
  Val x1424 = x1403[20]._super._super;
  Val x1425 = x1403[21]._super._super;
  Val x1426 = x1403[22]._super._super;
  Val x1427 = x1403[23]._super._super;
  Val x1428 = x1403[24]._super._super;
  Val x1429 = x1403[25]._super._super;
  Val x1430 = x1403[26]._super._super;
  Val x1431 = x1403[27]._super._super;
  Val x1432 = x1403[28]._super._super;
  Val x1433 = x1403[29]._super._super;
  Val x1434 = x1403[30]._super._super;
  Val x1435 = x1403[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1436 = x940.e._super;
  Val x1437 = x1436[0]._super._super;
  Val x1438 = x1436[1]._super._super;
  Val x1439 = x1436[2]._super._super;
  Val x1440 = x1436[3]._super._super;
  Val x1441 = x1436[4]._super._super;
  Val x1442 = x1436[5]._super._super;
  Val x1443 = x1436[6]._super._super;
  Val x1444 = x1436[7]._super._super;
  Val x1445 = x1436[8]._super._super;
  Val x1446 = x1436[9]._super._super;
  Val x1447 = x1436[10]._super._super;
  Val x1448 = x1436[11]._super._super;
  Val x1449 = x1436[12]._super._super;
  Val x1450 = x1436[13]._super._super;
  Val x1451 = x1436[14]._super._super;
  Val x1452 = x1436[15]._super._super;
  Val x1453 = x1436[16]._super._super;
  Val x1454 = x1436[17]._super._super;
  Val x1455 = x1436[18]._super._super;
  Val x1456 = x1436[19]._super._super;
  Val x1457 = x1436[20]._super._super;
  Val x1458 = x1436[21]._super._super;
  Val x1459 = x1436[22]._super._super;
  Val x1460 = x1436[23]._super._super;
  Val x1461 = x1436[24]._super._super;
  Val x1462 = x1436[25]._super._super;
  Val x1463 = x1436[26]._super._super;
  Val x1464 = x1436[27]._super._super;
  Val x1465 = x1436[28]._super._super;
  Val x1466 = x1436[29]._super._super;
  Val x1467 = x1436[30]._super._super;
  Val x1468 = x1436[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1469 = x926.w._super;
  Val x1470 = x1469[0]._super._super;
  Val x1471 = x1469[1]._super._super;
  Val x1472 = x1469[2]._super._super;
  Val x1473 = x1469[3]._super._super;
  Val x1474 = x1469[4]._super._super;
  Val x1475 = x1469[5]._super._super;
  Val x1476 = x1469[6]._super._super;
  Val x1477 = x1469[7]._super._super;
  Val x1478 = x1469[8]._super._super;
  Val x1479 = x1469[9]._super._super;
  Val x1480 = x1469[10]._super._super;
  Val x1481 = x1469[11]._super._super;
  Val x1482 = x1469[12]._super._super;
  Val x1483 = x1469[13]._super._super;
  Val x1484 = x1469[14]._super._super;
  Val x1485 = x1469[15]._super._super;
  Val x1486 = x1469[16]._super._super;
  Val x1487 = x1469[17]._super._super;
  Val x1488 = x1469[18]._super._super;
  Val x1489 = x1469[19]._super._super;
  Val x1490 = x1469[20]._super._super;
  Val x1491 = x1469[21]._super._super;
  Val x1492 = x1469[22]._super._super;
  Val x1493 = x1469[23]._super._super;
  Val x1494 = x1469[24]._super._super;
  Val x1495 = x1469[25]._super._super;
  Val x1496 = x1469[26]._super._super;
  Val x1497 = x1469[27]._super._super;
  Val x1498 = x1469[28]._super._super;
  Val x1499 = x1469[29]._super._super;
  Val x1500 = x1469[30]._super._super;
  Val x1501 = x1469[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1502 = x928.w._super;
  Val x1503 = x1502[0]._super._super;
  Val x1504 = x1502[1]._super._super;
  Val x1505 = x1502[2]._super._super;
  Val x1506 = x1502[3]._super._super;
  Val x1507 = x1502[4]._super._super;
  Val x1508 = x1502[5]._super._super;
  Val x1509 = x1502[6]._super._super;
  Val x1510 = x1502[7]._super._super;
  Val x1511 = x1502[8]._super._super;
  Val x1512 = x1502[9]._super._super;
  Val x1513 = x1502[10]._super._super;
  Val x1514 = x1502[11]._super._super;
  Val x1515 = x1502[12]._super._super;
  Val x1516 = x1502[13]._super._super;
  Val x1517 = x1502[14]._super._super;
  Val x1518 = x1502[15]._super._super;
  Val x1519 = x1502[16]._super._super;
  Val x1520 = x1502[17]._super._super;
  Val x1521 = x1502[18]._super._super;
  Val x1522 = x1502[19]._super._super;
  Val x1523 = x1502[20]._super._super;
  Val x1524 = x1502[21]._super._super;
  Val x1525 = x1502[22]._super._super;
  Val x1526 = x1502[23]._super._super;
  Val x1527 = x1502[24]._super._super;
  Val x1528 = x1502[25]._super._super;
  Val x1529 = x1502[26]._super._super;
  Val x1530 = x1502[27]._super._super;
  Val x1531 = x1502[28]._super._super;
  Val x1532 = x1502[29]._super._super;
  Val x1533 = x1502[30]._super._super;
  Val x1534 = x1502[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1535 = x930.w._super;
  Val x1536 = x1535[0]._super._super;
  Val x1537 = x1535[1]._super._super;
  Val x1538 = x1535[2]._super._super;
  Val x1539 = x1535[3]._super._super;
  Val x1540 = x1535[4]._super._super;
  Val x1541 = x1535[5]._super._super;
  Val x1542 = x1535[6]._super._super;
  Val x1543 = x1535[7]._super._super;
  Val x1544 = x1535[8]._super._super;
  Val x1545 = x1535[9]._super._super;
  Val x1546 = x1535[10]._super._super;
  Val x1547 = x1535[11]._super._super;
  Val x1548 = x1535[12]._super._super;
  Val x1549 = x1535[13]._super._super;
  Val x1550 = x1535[14]._super._super;
  Val x1551 = x1535[15]._super._super;
  Val x1552 = x1535[16]._super._super;
  Val x1553 = x1535[17]._super._super;
  Val x1554 = x1535[18]._super._super;
  Val x1555 = x1535[19]._super._super;
  Val x1556 = x1535[20]._super._super;
  Val x1557 = x1535[21]._super._super;
  Val x1558 = x1535[22]._super._super;
  Val x1559 = x1535[23]._super._super;
  Val x1560 = x1535[24]._super._super;
  Val x1561 = x1535[25]._super._super;
  Val x1562 = x1535[26]._super._super;
  Val x1563 = x1535[27]._super._super;
  Val x1564 = x1535[28]._super._super;
  Val x1565 = x1535[29]._super._super;
  Val x1566 = x1535[30]._super._super;
  Val x1567 = x1535[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1568 = x932.w._super;
  Val x1569 = x1568[0]._super._super;
  Val x1570 = x1568[1]._super._super;
  Val x1571 = x1568[2]._super._super;
  Val x1572 = x1568[3]._super._super;
  Val x1573 = x1568[4]._super._super;
  Val x1574 = x1568[5]._super._super;
  Val x1575 = x1568[6]._super._super;
  Val x1576 = x1568[7]._super._super;
  Val x1577 = x1568[8]._super._super;
  Val x1578 = x1568[9]._super._super;
  Val x1579 = x1568[10]._super._super;
  Val x1580 = x1568[11]._super._super;
  Val x1581 = x1568[12]._super._super;
  Val x1582 = x1568[13]._super._super;
  Val x1583 = x1568[14]._super._super;
  Val x1584 = x1568[15]._super._super;
  Val x1585 = x1568[16]._super._super;
  Val x1586 = x1568[17]._super._super;
  Val x1587 = x1568[18]._super._super;
  Val x1588 = x1568[19]._super._super;
  Val x1589 = x1568[20]._super._super;
  Val x1590 = x1568[21]._super._super;
  Val x1591 = x1568[22]._super._super;
  Val x1592 = x1568[23]._super._super;
  Val x1593 = x1568[24]._super._super;
  Val x1594 = x1568[25]._super._super;
  Val x1595 = x1568[26]._super._super;
  Val x1596 = x1568[27]._super._super;
  Val x1597 = x1568[28]._super._super;
  Val x1598 = x1568[29]._super._super;
  Val x1599 = x1568[30]._super._super;
  Val x1600 = x1568[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1601 = x934.w._super;
  Val x1602 = x1601[0]._super._super;
  Val x1603 = x1601[1]._super._super;
  Val x1604 = x1601[2]._super._super;
  Val x1605 = x1601[3]._super._super;
  Val x1606 = x1601[4]._super._super;
  Val x1607 = x1601[5]._super._super;
  Val x1608 = x1601[6]._super._super;
  Val x1609 = x1601[7]._super._super;
  Val x1610 = x1601[8]._super._super;
  Val x1611 = x1601[9]._super._super;
  Val x1612 = x1601[10]._super._super;
  Val x1613 = x1601[11]._super._super;
  Val x1614 = x1601[12]._super._super;
  Val x1615 = x1601[13]._super._super;
  Val x1616 = x1601[14]._super._super;
  Val x1617 = x1601[15]._super._super;
  Val x1618 = x1601[16]._super._super;
  Val x1619 = x1601[17]._super._super;
  Val x1620 = x1601[18]._super._super;
  Val x1621 = x1601[19]._super._super;
  Val x1622 = x1601[20]._super._super;
  Val x1623 = x1601[21]._super._super;
  Val x1624 = x1601[22]._super._super;
  Val x1625 = x1601[23]._super._super;
  Val x1626 = x1601[24]._super._super;
  Val x1627 = x1601[25]._super._super;
  Val x1628 = x1601[26]._super._super;
  Val x1629 = x1601[27]._super._super;
  Val x1630 = x1601[28]._super._super;
  Val x1631 = x1601[29]._super._super;
  Val x1632 = x1601[30]._super._super;
  Val x1633 = x1601[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1634 = x936.w._super;
  Val x1635 = x1634[0]._super._super;
  Val x1636 = x1634[1]._super._super;
  Val x1637 = x1634[2]._super._super;
  Val x1638 = x1634[3]._super._super;
  Val x1639 = x1634[4]._super._super;
  Val x1640 = x1634[5]._super._super;
  Val x1641 = x1634[6]._super._super;
  Val x1642 = x1634[7]._super._super;
  Val x1643 = x1634[8]._super._super;
  Val x1644 = x1634[9]._super._super;
  Val x1645 = x1634[10]._super._super;
  Val x1646 = x1634[11]._super._super;
  Val x1647 = x1634[12]._super._super;
  Val x1648 = x1634[13]._super._super;
  Val x1649 = x1634[14]._super._super;
  Val x1650 = x1634[15]._super._super;
  Val x1651 = x1634[16]._super._super;
  Val x1652 = x1634[17]._super._super;
  Val x1653 = x1634[18]._super._super;
  Val x1654 = x1634[19]._super._super;
  Val x1655 = x1634[20]._super._super;
  Val x1656 = x1634[21]._super._super;
  Val x1657 = x1634[22]._super._super;
  Val x1658 = x1634[23]._super._super;
  Val x1659 = x1634[24]._super._super;
  Val x1660 = x1634[25]._super._super;
  Val x1661 = x1634[26]._super._super;
  Val x1662 = x1634[27]._super._super;
  Val x1663 = x1634[28]._super._super;
  Val x1664 = x1634[29]._super._super;
  Val x1665 = x1634[30]._super._super;
  Val x1666 = x1634[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1667 = x938.w._super;
  Val x1668 = x1667[0]._super._super;
  Val x1669 = x1667[1]._super._super;
  Val x1670 = x1667[2]._super._super;
  Val x1671 = x1667[3]._super._super;
  Val x1672 = x1667[4]._super._super;
  Val x1673 = x1667[5]._super._super;
  Val x1674 = x1667[6]._super._super;
  Val x1675 = x1667[7]._super._super;
  Val x1676 = x1667[8]._super._super;
  Val x1677 = x1667[9]._super._super;
  Val x1678 = x1667[10]._super._super;
  Val x1679 = x1667[11]._super._super;
  Val x1680 = x1667[12]._super._super;
  Val x1681 = x1667[13]._super._super;
  Val x1682 = x1667[14]._super._super;
  Val x1683 = x1667[15]._super._super;
  Val x1684 = x1667[16]._super._super;
  Val x1685 = x1667[17]._super._super;
  Val x1686 = x1667[18]._super._super;
  Val x1687 = x1667[19]._super._super;
  Val x1688 = x1667[20]._super._super;
  Val x1689 = x1667[21]._super._super;
  Val x1690 = x1667[22]._super._super;
  Val x1691 = x1667[23]._super._super;
  Val x1692 = x1667[24]._super._super;
  Val x1693 = x1667[25]._super._super;
  Val x1694 = x1667[26]._super._super;
  Val x1695 = x1667[27]._super._super;
  Val x1696 = x1667[28]._super._super;
  Val x1697 = x1667[29]._super._super;
  Val x1698 = x1667[30]._super._super;
  Val x1699 = x1667[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x1700 = x940.w._super;
  Val x1701 = x1700[0]._super._super;
  Val x1702 = x1700[1]._super._super;
  Val x1703 = x1700[2]._super._super;
  Val x1704 = x1700[3]._super._super;
  Val x1705 = x1700[4]._super._super;
  Val x1706 = x1700[5]._super._super;
  Val x1707 = x1700[6]._super._super;
  Val x1708 = x1700[7]._super._super;
  Val x1709 = x1700[8]._super._super;
  Val x1710 = x1700[9]._super._super;
  Val x1711 = x1700[10]._super._super;
  Val x1712 = x1700[11]._super._super;
  Val x1713 = x1700[12]._super._super;
  Val x1714 = x1700[13]._super._super;
  Val x1715 = x1700[14]._super._super;
  Val x1716 = x1700[15]._super._super;
  Val x1717 = x1700[16]._super._super;
  Val x1718 = x1700[17]._super._super;
  Val x1719 = x1700[18]._super._super;
  Val x1720 = x1700[19]._super._super;
  Val x1721 = x1700[20]._super._super;
  Val x1722 = x1700[21]._super._super;
  Val x1723 = x1700[22]._super._super;
  Val x1724 = x1700[23]._super._super;
  Val x1725 = x1700[24]._super._super;
  Val x1726 = x1700[25]._super._super;
  Val x1727 = x1700[26]._super._super;
  Val x1728 = x1700[27]._super._super;
  Val x1729 = x1700[28]._super._super;
  Val x1730 = x1700[29]._super._super;
  Val x1731 = x1700[30]._super._super;
  Val x1732 = x1700[31]._super._super;
  // ShaCycle(zirgen/circuit/keccak2/top.zir:186)
  NondetRegStruct16Array x1733 = arg0.sflat;
  Val16Array x1734 = Val16Array{x1733[0]._super,
                                x1733[1]._super,
                                x1733[2]._super,
                                x1733[3]._super,
                                x1733[4]._super,
                                x1733[5]._super,
                                x1733[6]._super,
                                x1733[7]._super,
                                x1733[8]._super,
                                x1733[9]._super,
                                x1733[10]._super,
                                x1733[11]._super,
                                x1733[12]._super,
                                x1733[13]._super,
                                x1733[14]._super,
                                x1733[15]._super};
  // ShaCycle(zirgen/circuit/keccak2/top.zir:208)
  TopStateStruct x1735 = exec_TopState(
      ctx,
      Val800Array{
          x942,   x943,   x944,   x945,   x946,   x947,   x948,   x949,   x950,   x951,   x952,
          x953,   x954,   x955,   x956,   x957,   x958,   x959,   x960,   x961,   x962,   x963,
          x964,   x965,   x966,   x967,   x968,   x969,   x970,   x971,   x972,   x973,   x975,
          x976,   x977,   x978,   x979,   x980,   x981,   x982,   x983,   x984,   x985,   x986,
          x987,   x988,   x989,   x990,   x991,   x992,   x993,   x994,   x995,   x996,   x997,
          x998,   x999,   x1000,  x1001,  x1002,  x1003,  x1004,  x1005,  x1006,  x1008,  x1009,
          x1010,  x1011,  x1012,  x1013,  x1014,  x1015,  x1016,  x1017,  x1018,  x1019,  x1020,
          x1021,  x1022,  x1023,  x1024,  x1025,  x1026,  x1027,  x1028,  x1029,  x1030,  x1031,
          x1032,  x1033,  x1034,  x1035,  x1036,  x1037,  x1038,  x1039,  x1041,  x1042,  x1043,
          x1044,  x1045,  x1046,  x1047,  x1048,  x1049,  x1050,  x1051,  x1052,  x1053,  x1054,
          x1055,  x1056,  x1057,  x1058,  x1059,  x1060,  x1061,  x1062,  x1063,  x1064,  x1065,
          x1066,  x1067,  x1068,  x1069,  x1070,  x1071,  x1072,  x1074,  x1075,  x1076,  x1077,
          x1078,  x1079,  x1080,  x1081,  x1082,  x1083,  x1084,  x1085,  x1086,  x1087,  x1088,
          x1089,  x1090,  x1091,  x1092,  x1093,  x1094,  x1095,  x1096,  x1097,  x1098,  x1099,
          x1100,  x1101,  x1102,  x1103,  x1104,  x1105,  x1107,  x1108,  x1109,  x1110,  x1111,
          x1112,  x1113,  x1114,  x1115,  x1116,  x1117,  x1118,  x1119,  x1120,  x1121,  x1122,
          x1123,  x1124,  x1125,  x1126,  x1127,  x1128,  x1129,  x1130,  x1131,  x1132,  x1133,
          x1134,  x1135,  x1136,  x1137,  x1138,  x1140,  x1141,  x1142,  x1143,  x1144,  x1145,
          x1146,  x1147,  x1148,  x1149,  x1150,  x1151,  x1152,  x1153,  x1154,  x1155,  x1156,
          x1157,  x1158,  x1159,  x1160,  x1161,  x1162,  x1163,  x1164,  x1165,  x1166,  x1167,
          x1168,  x1169,  x1170,  x1171,  x1173,  x1174,  x1175,  x1176,  x1177,  x1178,  x1179,
          x1180,  x1181,  x1182,  x1183,  x1184,  x1185,  x1186,  x1187,  x1188,  x1189,  x1190,
          x1191,  x1192,  x1193,  x1194,  x1195,  x1196,  x1197,  x1198,  x1199,  x1200,  x1201,
          x1202,  x1203,  x1204,  x1206,  x1207,  x1208,  x1209,  x1210,  x1211,  x1212,  x1213,
          x1214,  x1215,  x1216,  x1217,  x1218,  x1219,  x1220,  x1221,  x1222,  x1223,  x1224,
          x1225,  x1226,  x1227,  x1228,  x1229,  x1230,  x1231,  x1232,  x1233,  x1234,  x1235,
          x1236,  x1237,  x1239,  x1240,  x1241,  x1242,  x1243,  x1244,  x1245,  x1246,  x1247,
          x1248,  x1249,  x1250,  x1251,  x1252,  x1253,  x1254,  x1255,  x1256,  x1257,  x1258,
          x1259,  x1260,  x1261,  x1262,  x1263,  x1264,  x1265,  x1266,  x1267,  x1268,  x1269,
          x1270,  x1272,  x1273,  x1274,  x1275,  x1276,  x1277,  x1278,  x1279,  x1280,  x1281,
          x1282,  x1283,  x1284,  x1285,  x1286,  x1287,  x1288,  x1289,  x1290,  x1291,  x1292,
          x1293,  x1294,  x1295,  x1296,  x1297,  x1298,  x1299,  x1300,  x1301,  x1302,  x1303,
          x1305,  x1306,  x1307,  x1308,  x1309,  x1310,  x1311,  x1312,  x1313,  x1314,  x1315,
          x1316,  x1317,  x1318,  x1319,  x1320,  x1321,  x1322,  x1323,  x1324,  x1325,  x1326,
          x1327,  x1328,  x1329,  x1330,  x1331,  x1332,  x1333,  x1334,  x1335,  x1336,  x1338,
          x1339,  x1340,  x1341,  x1342,  x1343,  x1344,  x1345,  x1346,  x1347,  x1348,  x1349,
          x1350,  x1351,  x1352,  x1353,  x1354,  x1355,  x1356,  x1357,  x1358,  x1359,  x1360,
          x1361,  x1362,  x1363,  x1364,  x1365,  x1366,  x1367,  x1368,  x1369,  x1371,  x1372,
          x1373,  x1374,  x1375,  x1376,  x1377,  x1378,  x1379,  x1380,  x1381,  x1382,  x1383,
          x1384,  x1385,  x1386,  x1387,  x1388,  x1389,  x1390,  x1391,  x1392,  x1393,  x1394,
          x1395,  x1396,  x1397,  x1398,  x1399,  x1400,  x1401,  x1402,  x1404,  x1405,  x1406,
          x1407,  x1408,  x1409,  x1410,  x1411,  x1412,  x1413,  x1414,  x1415,  x1416,  x1417,
          x1418,  x1419,  x1420,  x1421,  x1422,  x1423,  x1424,  x1425,  x1426,  x1427,  x1428,
          x1429,  x1430,  x1431,  x1432,  x1433,  x1434,  x1435,  x1437,  x1438,  x1439,  x1440,
          x1441,  x1442,  x1443,  x1444,  x1445,  x1446,  x1447,  x1448,  x1449,  x1450,  x1451,
          x1452,  x1453,  x1454,  x1455,  x1456,  x1457,  x1458,  x1459,  x1460,  x1461,  x1462,
          x1463,  x1464,  x1465,  x1466,  x1467,  x1468,  x1470,  x1471,  x1472,  x1473,  x1474,
          x1475,  x1476,  x1477,  x1478,  x1479,  x1480,  x1481,  x1482,  x1483,  x1484,  x1485,
          x1486,  x1487,  x1488,  x1489,  x1490,  x1491,  x1492,  x1493,  x1494,  x1495,  x1496,
          x1497,  x1498,  x1499,  x1500,  x1501,  x1503,  x1504,  x1505,  x1506,  x1507,  x1508,
          x1509,  x1510,  x1511,  x1512,  x1513,  x1514,  x1515,  x1516,  x1517,  x1518,  x1519,
          x1520,  x1521,  x1522,  x1523,  x1524,  x1525,  x1526,  x1527,  x1528,  x1529,  x1530,
          x1531,  x1532,  x1533,  x1534,  x1536,  x1537,  x1538,  x1539,  x1540,  x1541,  x1542,
          x1543,  x1544,  x1545,  x1546,  x1547,  x1548,  x1549,  x1550,  x1551,  x1552,  x1553,
          x1554,  x1555,  x1556,  x1557,  x1558,  x1559,  x1560,  x1561,  x1562,  x1563,  x1564,
          x1565,  x1566,  x1567,  x1569,  x1570,  x1571,  x1572,  x1573,  x1574,  x1575,  x1576,
          x1577,  x1578,  x1579,  x1580,  x1581,  x1582,  x1583,  x1584,  x1585,  x1586,  x1587,
          x1588,  x1589,  x1590,  x1591,  x1592,  x1593,  x1594,  x1595,  x1596,  x1597,  x1598,
          x1599,  x1600,  x1602,  x1603,  x1604,  x1605,  x1606,  x1607,  x1608,  x1609,  x1610,
          x1611,  x1612,  x1613,  x1614,  x1615,  x1616,  x1617,  x1618,  x1619,  x1620,  x1621,
          x1622,  x1623,  x1624,  x1625,  x1626,  x1627,  x1628,  x1629,  x1630,  x1631,  x1632,
          x1633,  x1635,  x1636,  x1637,  x1638,  x1639,  x1640,  x1641,  x1642,  x1643,  x1644,
          x1645,  x1646,  x1647,  x1648,  x1649,  x1650,  x1651,  x1652,  x1653,  x1654,  x1655,
          x1656,  x1657,  x1658,  x1659,  x1660,  x1661,  x1662,  x1663,  x1664,  x1665,  x1666,
          x1668,  x1669,  x1670,  x1671,  x1672,  x1673,  x1674,  x1675,  x1676,  x1677,  x1678,
          x1679,  x1680,  x1681,  x1682,  x1683,  x1684,  x1685,  x1686,  x1687,  x1688,  x1689,
          x1690,  x1691,  x1692,  x1693,  x1694,  x1695,  x1696,  x1697,  x1698,  x1699,  x1701,
          x1702,  x1703,  x1704,  x1705,  x1706,  x1707,  x1708,  x1709,  x1710,  x1711,  x1712,
          x1713,  x1714,  x1715,  x1716,  x1717,  x1718,  x1719,  x1720,  x1721,  x1722,  x1723,
          x1724,  x1725,  x1726,  x1727,  x1728,  x1729,  x1730,  x1731,  x1732,  Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      x921,
      x1734,
      LAYOUT_LOOKUP(layout4, _super));
  return x1735;
}

} // namespace risc0::impl
