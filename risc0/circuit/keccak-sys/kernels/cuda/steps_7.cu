#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"

namespace risc0::impl {

__device__ TopStateStruct exec_KeccakRound0(ExecContext& ctx,
                                            TopStateStruct arg0,
                                            TopStateStruct arg1,
                                            BoundLayout<KeccakRound0Layout> layout2) {
  // Log(<preamble>:22)
  // KeccakRound0(zirgen/circuit/keccak2/top.zir:95)
  INVOKE_EXTERN(ctx, log, "KeccakRound0", std::initializer_list<Val>{});
  // KeccakRound0(zirgen/circuit/keccak2/top.zir:94)
  TopStateBitsStruct800Array x3 = arg0.bits;
  TopStateBitsStruct800Array x4 = arg1.bits;
  // KeccakRound0(zirgen/circuit/keccak2/top.zir:96)
  Val x5 = x3[0]._super._super;
  Val x6 = x3[1]._super._super;
  Val x7 = x3[2]._super._super;
  Val x8 = x3[3]._super._super;
  Val x9 = x3[4]._super._super;
  Val x10 = x3[5]._super._super;
  Val x11 = x3[6]._super._super;
  Val x12 = x3[7]._super._super;
  Val x13 = x3[8]._super._super;
  Val x14 = x3[9]._super._super;
  Val x15 = x3[10]._super._super;
  Val x16 = x3[11]._super._super;
  Val x17 = x3[12]._super._super;
  Val x18 = x3[13]._super._super;
  Val x19 = x3[14]._super._super;
  Val x20 = x3[15]._super._super;
  Val x21 = x3[16]._super._super;
  Val x22 = x3[17]._super._super;
  Val x23 = x3[18]._super._super;
  Val x24 = x3[19]._super._super;
  Val x25 = x3[20]._super._super;
  Val x26 = x3[21]._super._super;
  Val x27 = x3[22]._super._super;
  Val x28 = x3[23]._super._super;
  Val x29 = x3[24]._super._super;
  Val x30 = x3[25]._super._super;
  Val x31 = x3[26]._super._super;
  Val x32 = x3[27]._super._super;
  Val x33 = x3[28]._super._super;
  Val x34 = x3[29]._super._super;
  Val x35 = x3[30]._super._super;
  Val x36 = x3[31]._super._super;
  Val x37 = x3[32]._super._super;
  Val x38 = x3[33]._super._super;
  Val x39 = x3[34]._super._super;
  Val x40 = x3[35]._super._super;
  Val x41 = x3[36]._super._super;
  Val x42 = x3[37]._super._super;
  Val x43 = x3[38]._super._super;
  Val x44 = x3[39]._super._super;
  Val x45 = x3[40]._super._super;
  Val x46 = x3[41]._super._super;
  Val x47 = x3[42]._super._super;
  Val x48 = x3[43]._super._super;
  Val x49 = x3[44]._super._super;
  Val x50 = x3[45]._super._super;
  Val x51 = x3[46]._super._super;
  Val x52 = x3[47]._super._super;
  Val x53 = x3[48]._super._super;
  Val x54 = x3[49]._super._super;
  Val x55 = x3[50]._super._super;
  Val x56 = x3[51]._super._super;
  Val x57 = x3[52]._super._super;
  Val x58 = x3[53]._super._super;
  Val x59 = x3[54]._super._super;
  Val x60 = x3[55]._super._super;
  Val x61 = x3[56]._super._super;
  Val x62 = x3[57]._super._super;
  Val x63 = x3[58]._super._super;
  Val x64 = x3[59]._super._super;
  Val x65 = x3[60]._super._super;
  Val x66 = x3[61]._super._super;
  Val x67 = x3[62]._super._super;
  Val x68 = x3[63]._super._super;
  Val x69 = x3[64]._super._super;
  Val x70 = x3[65]._super._super;
  Val x71 = x3[66]._super._super;
  Val x72 = x3[67]._super._super;
  Val x73 = x3[68]._super._super;
  Val x74 = x3[69]._super._super;
  Val x75 = x3[70]._super._super;
  Val x76 = x3[71]._super._super;
  Val x77 = x3[72]._super._super;
  Val x78 = x3[73]._super._super;
  Val x79 = x3[74]._super._super;
  Val x80 = x3[75]._super._super;
  Val x81 = x3[76]._super._super;
  Val x82 = x3[77]._super._super;
  Val x83 = x3[78]._super._super;
  Val x84 = x3[79]._super._super;
  Val x85 = x3[80]._super._super;
  Val x86 = x3[81]._super._super;
  Val x87 = x3[82]._super._super;
  Val x88 = x3[83]._super._super;
  Val x89 = x3[84]._super._super;
  Val x90 = x3[85]._super._super;
  Val x91 = x3[86]._super._super;
  Val x92 = x3[87]._super._super;
  Val x93 = x3[88]._super._super;
  Val x94 = x3[89]._super._super;
  Val x95 = x3[90]._super._super;
  Val x96 = x3[91]._super._super;
  Val x97 = x3[92]._super._super;
  Val x98 = x3[93]._super._super;
  Val x99 = x3[94]._super._super;
  Val x100 = x3[95]._super._super;
  Val x101 = x3[96]._super._super;
  Val x102 = x3[97]._super._super;
  Val x103 = x3[98]._super._super;
  Val x104 = x3[99]._super._super;
  Val x105 = x3[100]._super._super;
  Val x106 = x3[101]._super._super;
  Val x107 = x3[102]._super._super;
  Val x108 = x3[103]._super._super;
  Val x109 = x3[104]._super._super;
  Val x110 = x3[105]._super._super;
  Val x111 = x3[106]._super._super;
  Val x112 = x3[107]._super._super;
  Val x113 = x3[108]._super._super;
  Val x114 = x3[109]._super._super;
  Val x115 = x3[110]._super._super;
  Val x116 = x3[111]._super._super;
  Val x117 = x3[112]._super._super;
  Val x118 = x3[113]._super._super;
  Val x119 = x3[114]._super._super;
  Val x120 = x3[115]._super._super;
  Val x121 = x3[116]._super._super;
  Val x122 = x3[117]._super._super;
  Val x123 = x3[118]._super._super;
  Val x124 = x3[119]._super._super;
  Val x125 = x3[120]._super._super;
  Val x126 = x3[121]._super._super;
  Val x127 = x3[122]._super._super;
  Val x128 = x3[123]._super._super;
  Val x129 = x3[124]._super._super;
  Val x130 = x3[125]._super._super;
  Val x131 = x3[126]._super._super;
  Val x132 = x3[127]._super._super;
  Val x133 = x3[128]._super._super;
  Val x134 = x3[129]._super._super;
  Val x135 = x3[130]._super._super;
  Val x136 = x3[131]._super._super;
  Val x137 = x3[132]._super._super;
  Val x138 = x3[133]._super._super;
  Val x139 = x3[134]._super._super;
  Val x140 = x3[135]._super._super;
  Val x141 = x3[136]._super._super;
  Val x142 = x3[137]._super._super;
  Val x143 = x3[138]._super._super;
  Val x144 = x3[139]._super._super;
  Val x145 = x3[140]._super._super;
  Val x146 = x3[141]._super._super;
  Val x147 = x3[142]._super._super;
  Val x148 = x3[143]._super._super;
  Val x149 = x3[144]._super._super;
  Val x150 = x3[145]._super._super;
  Val x151 = x3[146]._super._super;
  Val x152 = x3[147]._super._super;
  Val x153 = x3[148]._super._super;
  Val x154 = x3[149]._super._super;
  Val x155 = x3[150]._super._super;
  Val x156 = x3[151]._super._super;
  Val x157 = x3[152]._super._super;
  Val x158 = x3[153]._super._super;
  Val x159 = x3[154]._super._super;
  Val x160 = x3[155]._super._super;
  Val x161 = x3[156]._super._super;
  Val x162 = x3[157]._super._super;
  Val x163 = x3[158]._super._super;
  Val x164 = x3[159]._super._super;
  Val x165 = x3[160]._super._super;
  Val x166 = x3[161]._super._super;
  Val x167 = x3[162]._super._super;
  Val x168 = x3[163]._super._super;
  Val x169 = x3[164]._super._super;
  Val x170 = x3[165]._super._super;
  Val x171 = x3[166]._super._super;
  Val x172 = x3[167]._super._super;
  Val x173 = x3[168]._super._super;
  Val x174 = x3[169]._super._super;
  Val x175 = x3[170]._super._super;
  Val x176 = x3[171]._super._super;
  Val x177 = x3[172]._super._super;
  Val x178 = x3[173]._super._super;
  Val x179 = x3[174]._super._super;
  Val x180 = x3[175]._super._super;
  Val x181 = x3[176]._super._super;
  Val x182 = x3[177]._super._super;
  Val x183 = x3[178]._super._super;
  Val x184 = x3[179]._super._super;
  Val x185 = x3[180]._super._super;
  Val x186 = x3[181]._super._super;
  Val x187 = x3[182]._super._super;
  Val x188 = x3[183]._super._super;
  Val x189 = x3[184]._super._super;
  Val x190 = x3[185]._super._super;
  Val x191 = x3[186]._super._super;
  Val x192 = x3[187]._super._super;
  Val x193 = x3[188]._super._super;
  Val x194 = x3[189]._super._super;
  Val x195 = x3[190]._super._super;
  Val x196 = x3[191]._super._super;
  Val x197 = x3[192]._super._super;
  Val x198 = x3[193]._super._super;
  Val x199 = x3[194]._super._super;
  Val x200 = x3[195]._super._super;
  Val x201 = x3[196]._super._super;
  Val x202 = x3[197]._super._super;
  Val x203 = x3[198]._super._super;
  Val x204 = x3[199]._super._super;
  Val x205 = x3[200]._super._super;
  Val x206 = x3[201]._super._super;
  Val x207 = x3[202]._super._super;
  Val x208 = x3[203]._super._super;
  Val x209 = x3[204]._super._super;
  Val x210 = x3[205]._super._super;
  Val x211 = x3[206]._super._super;
  Val x212 = x3[207]._super._super;
  Val x213 = x3[208]._super._super;
  Val x214 = x3[209]._super._super;
  Val x215 = x3[210]._super._super;
  Val x216 = x3[211]._super._super;
  Val x217 = x3[212]._super._super;
  Val x218 = x3[213]._super._super;
  Val x219 = x3[214]._super._super;
  Val x220 = x3[215]._super._super;
  Val x221 = x3[216]._super._super;
  Val x222 = x3[217]._super._super;
  Val x223 = x3[218]._super._super;
  Val x224 = x3[219]._super._super;
  Val x225 = x3[220]._super._super;
  Val x226 = x3[221]._super._super;
  Val x227 = x3[222]._super._super;
  Val x228 = x3[223]._super._super;
  Val x229 = x3[224]._super._super;
  Val x230 = x3[225]._super._super;
  Val x231 = x3[226]._super._super;
  Val x232 = x3[227]._super._super;
  Val x233 = x3[228]._super._super;
  Val x234 = x3[229]._super._super;
  Val x235 = x3[230]._super._super;
  Val x236 = x3[231]._super._super;
  Val x237 = x3[232]._super._super;
  Val x238 = x3[233]._super._super;
  Val x239 = x3[234]._super._super;
  Val x240 = x3[235]._super._super;
  Val x241 = x3[236]._super._super;
  Val x242 = x3[237]._super._super;
  Val x243 = x3[238]._super._super;
  Val x244 = x3[239]._super._super;
  Val x245 = x3[240]._super._super;
  Val x246 = x3[241]._super._super;
  Val x247 = x3[242]._super._super;
  Val x248 = x3[243]._super._super;
  Val x249 = x3[244]._super._super;
  Val x250 = x3[245]._super._super;
  Val x251 = x3[246]._super._super;
  Val x252 = x3[247]._super._super;
  Val x253 = x3[248]._super._super;
  Val x254 = x3[249]._super._super;
  Val x255 = x3[250]._super._super;
  Val x256 = x3[251]._super._super;
  Val x257 = x3[252]._super._super;
  Val x258 = x3[253]._super._super;
  Val x259 = x3[254]._super._super;
  Val x260 = x3[255]._super._super;
  Val x261 = x3[256]._super._super;
  Val x262 = x3[257]._super._super;
  Val x263 = x3[258]._super._super;
  Val x264 = x3[259]._super._super;
  Val x265 = x3[260]._super._super;
  Val x266 = x3[261]._super._super;
  Val x267 = x3[262]._super._super;
  Val x268 = x3[263]._super._super;
  Val x269 = x3[264]._super._super;
  Val x270 = x3[265]._super._super;
  Val x271 = x3[266]._super._super;
  Val x272 = x3[267]._super._super;
  Val x273 = x3[268]._super._super;
  Val x274 = x3[269]._super._super;
  Val x275 = x3[270]._super._super;
  Val x276 = x3[271]._super._super;
  Val x277 = x3[272]._super._super;
  Val x278 = x3[273]._super._super;
  Val x279 = x3[274]._super._super;
  Val x280 = x3[275]._super._super;
  Val x281 = x3[276]._super._super;
  Val x282 = x3[277]._super._super;
  Val x283 = x3[278]._super._super;
  Val x284 = x3[279]._super._super;
  Val x285 = x3[280]._super._super;
  Val x286 = x3[281]._super._super;
  Val x287 = x3[282]._super._super;
  Val x288 = x3[283]._super._super;
  Val x289 = x3[284]._super._super;
  Val x290 = x3[285]._super._super;
  Val x291 = x3[286]._super._super;
  Val x292 = x3[287]._super._super;
  Val x293 = x3[288]._super._super;
  Val x294 = x3[289]._super._super;
  Val x295 = x3[290]._super._super;
  Val x296 = x3[291]._super._super;
  Val x297 = x3[292]._super._super;
  Val x298 = x3[293]._super._super;
  Val x299 = x3[294]._super._super;
  Val x300 = x3[295]._super._super;
  Val x301 = x3[296]._super._super;
  Val x302 = x3[297]._super._super;
  Val x303 = x3[298]._super._super;
  Val x304 = x3[299]._super._super;
  Val x305 = x3[300]._super._super;
  Val x306 = x3[301]._super._super;
  Val x307 = x3[302]._super._super;
  Val x308 = x3[303]._super._super;
  Val x309 = x3[304]._super._super;
  Val x310 = x3[305]._super._super;
  Val x311 = x3[306]._super._super;
  Val x312 = x3[307]._super._super;
  Val x313 = x3[308]._super._super;
  Val x314 = x3[309]._super._super;
  Val x315 = x3[310]._super._super;
  Val x316 = x3[311]._super._super;
  Val x317 = x3[312]._super._super;
  Val x318 = x3[313]._super._super;
  Val x319 = x3[314]._super._super;
  Val x320 = x3[315]._super._super;
  Val x321 = x3[316]._super._super;
  Val x322 = x3[317]._super._super;
  Val x323 = x3[318]._super._super;
  Val x324 = x3[319]._super._super;
  Val x325 = x3[320]._super._super;
  Val x326 = x3[321]._super._super;
  Val x327 = x3[322]._super._super;
  Val x328 = x3[323]._super._super;
  Val x329 = x3[324]._super._super;
  Val x330 = x3[325]._super._super;
  Val x331 = x3[326]._super._super;
  Val x332 = x3[327]._super._super;
  Val x333 = x3[328]._super._super;
  Val x334 = x3[329]._super._super;
  Val x335 = x3[330]._super._super;
  Val x336 = x3[331]._super._super;
  Val x337 = x3[332]._super._super;
  Val x338 = x3[333]._super._super;
  Val x339 = x3[334]._super._super;
  Val x340 = x3[335]._super._super;
  Val x341 = x3[336]._super._super;
  Val x342 = x3[337]._super._super;
  Val x343 = x3[338]._super._super;
  Val x344 = x3[339]._super._super;
  Val x345 = x3[340]._super._super;
  Val x346 = x3[341]._super._super;
  Val x347 = x3[342]._super._super;
  Val x348 = x3[343]._super._super;
  Val x349 = x3[344]._super._super;
  Val x350 = x3[345]._super._super;
  Val x351 = x3[346]._super._super;
  Val x352 = x3[347]._super._super;
  Val x353 = x3[348]._super._super;
  Val x354 = x3[349]._super._super;
  Val x355 = x3[350]._super._super;
  Val x356 = x3[351]._super._super;
  Val x357 = x3[352]._super._super;
  Val x358 = x3[353]._super._super;
  Val x359 = x3[354]._super._super;
  Val x360 = x3[355]._super._super;
  Val x361 = x3[356]._super._super;
  Val x362 = x3[357]._super._super;
  Val x363 = x3[358]._super._super;
  Val x364 = x3[359]._super._super;
  Val x365 = x3[360]._super._super;
  Val x366 = x3[361]._super._super;
  Val x367 = x3[362]._super._super;
  Val x368 = x3[363]._super._super;
  Val x369 = x3[364]._super._super;
  Val x370 = x3[365]._super._super;
  Val x371 = x3[366]._super._super;
  Val x372 = x3[367]._super._super;
  Val x373 = x3[368]._super._super;
  Val x374 = x3[369]._super._super;
  Val x375 = x3[370]._super._super;
  Val x376 = x3[371]._super._super;
  Val x377 = x3[372]._super._super;
  Val x378 = x3[373]._super._super;
  Val x379 = x3[374]._super._super;
  Val x380 = x3[375]._super._super;
  Val x381 = x3[376]._super._super;
  Val x382 = x3[377]._super._super;
  Val x383 = x3[378]._super._super;
  Val x384 = x3[379]._super._super;
  Val x385 = x3[380]._super._super;
  Val x386 = x3[381]._super._super;
  Val x387 = x3[382]._super._super;
  Val x388 = x3[383]._super._super;
  Val x389 = x3[384]._super._super;
  Val x390 = x3[385]._super._super;
  Val x391 = x3[386]._super._super;
  Val x392 = x3[387]._super._super;
  Val x393 = x3[388]._super._super;
  Val x394 = x3[389]._super._super;
  Val x395 = x3[390]._super._super;
  Val x396 = x3[391]._super._super;
  Val x397 = x3[392]._super._super;
  Val x398 = x3[393]._super._super;
  Val x399 = x3[394]._super._super;
  Val x400 = x3[395]._super._super;
  Val x401 = x3[396]._super._super;
  Val x402 = x3[397]._super._super;
  Val x403 = x3[398]._super._super;
  Val x404 = x3[399]._super._super;
  Val x405 = x3[400]._super._super;
  Val x406 = x3[401]._super._super;
  Val x407 = x3[402]._super._super;
  Val x408 = x3[403]._super._super;
  Val x409 = x3[404]._super._super;
  Val x410 = x3[405]._super._super;
  Val x411 = x3[406]._super._super;
  Val x412 = x3[407]._super._super;
  Val x413 = x3[408]._super._super;
  Val x414 = x3[409]._super._super;
  Val x415 = x3[410]._super._super;
  Val x416 = x3[411]._super._super;
  Val x417 = x3[412]._super._super;
  Val x418 = x3[413]._super._super;
  Val x419 = x3[414]._super._super;
  Val x420 = x3[415]._super._super;
  Val x421 = x3[416]._super._super;
  Val x422 = x3[417]._super._super;
  Val x423 = x3[418]._super._super;
  Val x424 = x3[419]._super._super;
  Val x425 = x3[420]._super._super;
  Val x426 = x3[421]._super._super;
  Val x427 = x3[422]._super._super;
  Val x428 = x3[423]._super._super;
  Val x429 = x3[424]._super._super;
  Val x430 = x3[425]._super._super;
  Val x431 = x3[426]._super._super;
  Val x432 = x3[427]._super._super;
  Val x433 = x3[428]._super._super;
  Val x434 = x3[429]._super._super;
  Val x435 = x3[430]._super._super;
  Val x436 = x3[431]._super._super;
  Val x437 = x3[432]._super._super;
  Val x438 = x3[433]._super._super;
  Val x439 = x3[434]._super._super;
  Val x440 = x3[435]._super._super;
  Val x441 = x3[436]._super._super;
  Val x442 = x3[437]._super._super;
  Val x443 = x3[438]._super._super;
  Val x444 = x3[439]._super._super;
  Val x445 = x3[440]._super._super;
  Val x446 = x3[441]._super._super;
  Val x447 = x3[442]._super._super;
  Val x448 = x3[443]._super._super;
  Val x449 = x3[444]._super._super;
  Val x450 = x3[445]._super._super;
  Val x451 = x3[446]._super._super;
  Val x452 = x3[447]._super._super;
  Val x453 = x3[448]._super._super;
  Val x454 = x3[449]._super._super;
  Val x455 = x3[450]._super._super;
  Val x456 = x3[451]._super._super;
  Val x457 = x3[452]._super._super;
  Val x458 = x3[453]._super._super;
  Val x459 = x3[454]._super._super;
  Val x460 = x3[455]._super._super;
  Val x461 = x3[456]._super._super;
  Val x462 = x3[457]._super._super;
  Val x463 = x3[458]._super._super;
  Val x464 = x3[459]._super._super;
  Val x465 = x3[460]._super._super;
  Val x466 = x3[461]._super._super;
  Val x467 = x3[462]._super._super;
  Val x468 = x3[463]._super._super;
  Val x469 = x3[464]._super._super;
  Val x470 = x3[465]._super._super;
  Val x471 = x3[466]._super._super;
  Val x472 = x3[467]._super._super;
  Val x473 = x3[468]._super._super;
  Val x474 = x3[469]._super._super;
  Val x475 = x3[470]._super._super;
  Val x476 = x3[471]._super._super;
  Val x477 = x3[472]._super._super;
  Val x478 = x3[473]._super._super;
  Val x479 = x3[474]._super._super;
  Val x480 = x3[475]._super._super;
  Val x481 = x3[476]._super._super;
  Val x482 = x3[477]._super._super;
  Val x483 = x3[478]._super._super;
  Val x484 = x3[479]._super._super;
  Val x485 = x3[480]._super._super;
  Val x486 = x3[481]._super._super;
  Val x487 = x3[482]._super._super;
  Val x488 = x3[483]._super._super;
  Val x489 = x3[484]._super._super;
  Val x490 = x3[485]._super._super;
  Val x491 = x3[486]._super._super;
  Val x492 = x3[487]._super._super;
  Val x493 = x3[488]._super._super;
  Val x494 = x3[489]._super._super;
  Val x495 = x3[490]._super._super;
  Val x496 = x3[491]._super._super;
  Val x497 = x3[492]._super._super;
  Val x498 = x3[493]._super._super;
  Val x499 = x3[494]._super._super;
  Val x500 = x3[495]._super._super;
  Val x501 = x3[496]._super._super;
  Val x502 = x3[497]._super._super;
  Val x503 = x3[498]._super._super;
  Val x504 = x3[499]._super._super;
  Val x505 = x3[500]._super._super;
  Val x506 = x3[501]._super._super;
  Val x507 = x3[502]._super._super;
  Val x508 = x3[503]._super._super;
  Val x509 = x3[504]._super._super;
  Val x510 = x3[505]._super._super;
  Val x511 = x3[506]._super._super;
  Val x512 = x3[507]._super._super;
  Val x513 = x3[508]._super._super;
  Val x514 = x3[509]._super._super;
  Val x515 = x3[510]._super._super;
  Val x516 = x3[511]._super._super;
  Val x517 = x3[512]._super._super;
  Val x518 = x3[513]._super._super;
  Val x519 = x3[514]._super._super;
  Val x520 = x3[515]._super._super;
  Val x521 = x3[516]._super._super;
  Val x522 = x3[517]._super._super;
  Val x523 = x3[518]._super._super;
  Val x524 = x3[519]._super._super;
  Val x525 = x3[520]._super._super;
  Val x526 = x3[521]._super._super;
  Val x527 = x3[522]._super._super;
  Val x528 = x3[523]._super._super;
  Val x529 = x3[524]._super._super;
  Val x530 = x3[525]._super._super;
  Val x531 = x3[526]._super._super;
  Val x532 = x3[527]._super._super;
  Val x533 = x3[528]._super._super;
  Val x534 = x3[529]._super._super;
  Val x535 = x3[530]._super._super;
  Val x536 = x3[531]._super._super;
  Val x537 = x3[532]._super._super;
  Val x538 = x3[533]._super._super;
  Val x539 = x3[534]._super._super;
  Val x540 = x3[535]._super._super;
  Val x541 = x3[536]._super._super;
  Val x542 = x3[537]._super._super;
  Val x543 = x3[538]._super._super;
  Val x544 = x3[539]._super._super;
  Val x545 = x3[540]._super._super;
  Val x546 = x3[541]._super._super;
  Val x547 = x3[542]._super._super;
  Val x548 = x3[543]._super._super;
  Val x549 = x3[544]._super._super;
  Val x550 = x3[545]._super._super;
  Val x551 = x3[546]._super._super;
  Val x552 = x3[547]._super._super;
  Val x553 = x3[548]._super._super;
  Val x554 = x3[549]._super._super;
  Val x555 = x3[550]._super._super;
  Val x556 = x3[551]._super._super;
  Val x557 = x3[552]._super._super;
  Val x558 = x3[553]._super._super;
  Val x559 = x3[554]._super._super;
  Val x560 = x3[555]._super._super;
  Val x561 = x3[556]._super._super;
  Val x562 = x3[557]._super._super;
  Val x563 = x3[558]._super._super;
  Val x564 = x3[559]._super._super;
  Val x565 = x3[560]._super._super;
  Val x566 = x3[561]._super._super;
  Val x567 = x3[562]._super._super;
  Val x568 = x3[563]._super._super;
  Val x569 = x3[564]._super._super;
  Val x570 = x3[565]._super._super;
  Val x571 = x3[566]._super._super;
  Val x572 = x3[567]._super._super;
  Val x573 = x3[568]._super._super;
  Val x574 = x3[569]._super._super;
  Val x575 = x3[570]._super._super;
  Val x576 = x3[571]._super._super;
  Val x577 = x3[572]._super._super;
  Val x578 = x3[573]._super._super;
  Val x579 = x3[574]._super._super;
  Val x580 = x3[575]._super._super;
  Val x581 = x3[576]._super._super;
  Val x582 = x3[577]._super._super;
  Val x583 = x3[578]._super._super;
  Val x584 = x3[579]._super._super;
  Val x585 = x3[580]._super._super;
  Val x586 = x3[581]._super._super;
  Val x587 = x3[582]._super._super;
  Val x588 = x3[583]._super._super;
  Val x589 = x3[584]._super._super;
  Val x590 = x3[585]._super._super;
  Val x591 = x3[586]._super._super;
  Val x592 = x3[587]._super._super;
  Val x593 = x3[588]._super._super;
  Val x594 = x3[589]._super._super;
  Val x595 = x3[590]._super._super;
  Val x596 = x3[591]._super._super;
  Val x597 = x3[592]._super._super;
  Val x598 = x3[593]._super._super;
  Val x599 = x3[594]._super._super;
  Val x600 = x3[595]._super._super;
  Val x601 = x3[596]._super._super;
  Val x602 = x3[597]._super._super;
  Val x603 = x3[598]._super._super;
  Val x604 = x3[599]._super._super;
  Val x605 = x3[600]._super._super;
  Val x606 = x3[601]._super._super;
  Val x607 = x3[602]._super._super;
  Val x608 = x3[603]._super._super;
  Val x609 = x3[604]._super._super;
  Val x610 = x3[605]._super._super;
  Val x611 = x3[606]._super._super;
  Val x612 = x3[607]._super._super;
  Val x613 = x3[608]._super._super;
  Val x614 = x3[609]._super._super;
  Val x615 = x3[610]._super._super;
  Val x616 = x3[611]._super._super;
  Val x617 = x3[612]._super._super;
  Val x618 = x3[613]._super._super;
  Val x619 = x3[614]._super._super;
  Val x620 = x3[615]._super._super;
  Val x621 = x3[616]._super._super;
  Val x622 = x3[617]._super._super;
  Val x623 = x3[618]._super._super;
  Val x624 = x3[619]._super._super;
  Val x625 = x3[620]._super._super;
  Val x626 = x3[621]._super._super;
  Val x627 = x3[622]._super._super;
  Val x628 = x3[623]._super._super;
  Val x629 = x3[624]._super._super;
  Val x630 = x3[625]._super._super;
  Val x631 = x3[626]._super._super;
  Val x632 = x3[627]._super._super;
  Val x633 = x3[628]._super._super;
  Val x634 = x3[629]._super._super;
  Val x635 = x3[630]._super._super;
  Val x636 = x3[631]._super._super;
  Val x637 = x3[632]._super._super;
  Val x638 = x3[633]._super._super;
  Val x639 = x3[634]._super._super;
  Val x640 = x3[635]._super._super;
  Val x641 = x3[636]._super._super;
  Val x642 = x3[637]._super._super;
  Val x643 = x3[638]._super._super;
  Val x644 = x3[639]._super._super;
  Val x645 = x3[640]._super._super;
  Val x646 = x3[641]._super._super;
  Val x647 = x3[642]._super._super;
  Val x648 = x3[643]._super._super;
  Val x649 = x3[644]._super._super;
  Val x650 = x3[645]._super._super;
  Val x651 = x3[646]._super._super;
  Val x652 = x3[647]._super._super;
  Val x653 = x3[648]._super._super;
  Val x654 = x3[649]._super._super;
  Val x655 = x3[650]._super._super;
  Val x656 = x3[651]._super._super;
  Val x657 = x3[652]._super._super;
  Val x658 = x3[653]._super._super;
  Val x659 = x3[654]._super._super;
  Val x660 = x3[655]._super._super;
  Val x661 = x3[656]._super._super;
  Val x662 = x3[657]._super._super;
  Val x663 = x3[658]._super._super;
  Val x664 = x3[659]._super._super;
  Val x665 = x3[660]._super._super;
  Val x666 = x3[661]._super._super;
  Val x667 = x3[662]._super._super;
  Val x668 = x3[663]._super._super;
  Val x669 = x3[664]._super._super;
  Val x670 = x3[665]._super._super;
  Val x671 = x3[666]._super._super;
  Val x672 = x3[667]._super._super;
  Val x673 = x3[668]._super._super;
  Val x674 = x3[669]._super._super;
  Val x675 = x3[670]._super._super;
  Val x676 = x3[671]._super._super;
  Val x677 = x3[672]._super._super;
  Val x678 = x3[673]._super._super;
  Val x679 = x3[674]._super._super;
  Val x680 = x3[675]._super._super;
  Val x681 = x3[676]._super._super;
  Val x682 = x3[677]._super._super;
  Val x683 = x3[678]._super._super;
  Val x684 = x3[679]._super._super;
  Val x685 = x3[680]._super._super;
  Val x686 = x3[681]._super._super;
  Val x687 = x3[682]._super._super;
  Val x688 = x3[683]._super._super;
  Val x689 = x3[684]._super._super;
  Val x690 = x3[685]._super._super;
  Val x691 = x3[686]._super._super;
  Val x692 = x3[687]._super._super;
  Val x693 = x3[688]._super._super;
  Val x694 = x3[689]._super._super;
  Val x695 = x3[690]._super._super;
  Val x696 = x3[691]._super._super;
  Val x697 = x3[692]._super._super;
  Val x698 = x3[693]._super._super;
  Val x699 = x3[694]._super._super;
  Val x700 = x3[695]._super._super;
  Val x701 = x3[696]._super._super;
  Val x702 = x3[697]._super._super;
  Val x703 = x3[698]._super._super;
  Val x704 = x3[699]._super._super;
  Val x705 = x3[700]._super._super;
  Val x706 = x3[701]._super._super;
  Val x707 = x3[702]._super._super;
  Val x708 = x3[703]._super._super;
  Val x709 = x3[704]._super._super;
  Val x710 = x3[705]._super._super;
  Val x711 = x3[706]._super._super;
  Val x712 = x3[707]._super._super;
  Val x713 = x3[708]._super._super;
  Val x714 = x3[709]._super._super;
  Val x715 = x3[710]._super._super;
  Val x716 = x3[711]._super._super;
  Val x717 = x3[712]._super._super;
  Val x718 = x3[713]._super._super;
  Val x719 = x3[714]._super._super;
  Val x720 = x3[715]._super._super;
  Val x721 = x3[716]._super._super;
  Val x722 = x3[717]._super._super;
  Val x723 = x3[718]._super._super;
  Val x724 = x3[719]._super._super;
  Val x725 = x3[720]._super._super;
  Val x726 = x3[721]._super._super;
  Val x727 = x3[722]._super._super;
  Val x728 = x3[723]._super._super;
  Val x729 = x3[724]._super._super;
  Val x730 = x3[725]._super._super;
  Val x731 = x3[726]._super._super;
  Val x732 = x3[727]._super._super;
  Val x733 = x3[728]._super._super;
  Val x734 = x3[729]._super._super;
  Val x735 = x3[730]._super._super;
  Val x736 = x3[731]._super._super;
  Val x737 = x3[732]._super._super;
  Val x738 = x3[733]._super._super;
  Val x739 = x3[734]._super._super;
  Val x740 = x3[735]._super._super;
  Val x741 = x3[736]._super._super;
  Val x742 = x3[737]._super._super;
  Val x743 = x3[738]._super._super;
  Val x744 = x3[739]._super._super;
  Val x745 = x3[740]._super._super;
  Val x746 = x3[741]._super._super;
  Val x747 = x3[742]._super._super;
  Val x748 = x3[743]._super._super;
  Val x749 = x3[744]._super._super;
  Val x750 = x3[745]._super._super;
  Val x751 = x3[746]._super._super;
  Val x752 = x3[747]._super._super;
  Val x753 = x3[748]._super._super;
  Val x754 = x3[749]._super._super;
  Val x755 = x3[750]._super._super;
  Val x756 = x3[751]._super._super;
  Val x757 = x3[752]._super._super;
  Val x758 = x3[753]._super._super;
  Val x759 = x3[754]._super._super;
  Val x760 = x3[755]._super._super;
  Val x761 = x3[756]._super._super;
  Val x762 = x3[757]._super._super;
  Val x763 = x3[758]._super._super;
  Val x764 = x3[759]._super._super;
  Val x765 = x3[760]._super._super;
  Val x766 = x3[761]._super._super;
  Val x767 = x3[762]._super._super;
  Val x768 = x3[763]._super._super;
  Val x769 = x3[764]._super._super;
  Val x770 = x3[765]._super._super;
  Val x771 = x3[766]._super._super;
  Val x772 = x3[767]._super._super;
  Val x773 = x3[768]._super._super;
  Val x774 = x3[769]._super._super;
  Val x775 = x3[770]._super._super;
  Val x776 = x3[771]._super._super;
  Val x777 = x3[772]._super._super;
  Val x778 = x3[773]._super._super;
  Val x779 = x3[774]._super._super;
  Val x780 = x3[775]._super._super;
  Val x781 = x3[776]._super._super;
  Val x782 = x3[777]._super._super;
  Val x783 = x3[778]._super._super;
  Val x784 = x3[779]._super._super;
  Val x785 = x3[780]._super._super;
  Val x786 = x3[781]._super._super;
  Val x787 = x3[782]._super._super;
  Val x788 = x3[783]._super._super;
  Val x789 = x3[784]._super._super;
  Val x790 = x3[785]._super._super;
  Val x791 = x3[786]._super._super;
  Val x792 = x3[787]._super._super;
  Val x793 = x3[788]._super._super;
  Val x794 = x3[789]._super._super;
  Val x795 = x3[790]._super._super;
  Val x796 = x3[791]._super._super;
  Val x797 = x3[792]._super._super;
  Val x798 = x3[793]._super._super;
  Val x799 = x3[794]._super._super;
  Val x800 = x3[795]._super._super;
  Val x801 = x3[796]._super._super;
  Val x802 = x3[797]._super._super;
  Val x803 = x3[798]._super._super;
  Val x804 = x3[799]._super._super;
  Val x805 = x4[0]._super._super;
  Val x806 = x4[1]._super._super;
  Val x807 = x4[2]._super._super;
  Val x808 = x4[3]._super._super;
  Val x809 = x4[4]._super._super;
  Val x810 = x4[5]._super._super;
  Val x811 = x4[6]._super._super;
  Val x812 = x4[7]._super._super;
  Val x813 = x4[8]._super._super;
  Val x814 = x4[9]._super._super;
  Val x815 = x4[10]._super._super;
  Val x816 = x4[11]._super._super;
  Val x817 = x4[12]._super._super;
  Val x818 = x4[13]._super._super;
  Val x819 = x4[14]._super._super;
  Val x820 = x4[15]._super._super;
  Val x821 = x4[16]._super._super;
  Val x822 = x4[17]._super._super;
  Val x823 = x4[18]._super._super;
  Val x824 = x4[19]._super._super;
  Val x825 = x4[20]._super._super;
  Val x826 = x4[21]._super._super;
  Val x827 = x4[22]._super._super;
  Val x828 = x4[23]._super._super;
  Val x829 = x4[24]._super._super;
  Val x830 = x4[25]._super._super;
  Val x831 = x4[26]._super._super;
  Val x832 = x4[27]._super._super;
  Val x833 = x4[28]._super._super;
  Val x834 = x4[29]._super._super;
  Val x835 = x4[30]._super._super;
  Val x836 = x4[31]._super._super;
  Val x837 = x4[32]._super._super;
  Val x838 = x4[33]._super._super;
  Val x839 = x4[34]._super._super;
  Val x840 = x4[35]._super._super;
  Val x841 = x4[36]._super._super;
  Val x842 = x4[37]._super._super;
  Val x843 = x4[38]._super._super;
  Val x844 = x4[39]._super._super;
  Val x845 = x4[40]._super._super;
  Val x846 = x4[41]._super._super;
  Val x847 = x4[42]._super._super;
  Val x848 = x4[43]._super._super;
  Val x849 = x4[44]._super._super;
  Val x850 = x4[45]._super._super;
  Val x851 = x4[46]._super._super;
  Val x852 = x4[47]._super._super;
  Val x853 = x4[48]._super._super;
  Val x854 = x4[49]._super._super;
  Val x855 = x4[50]._super._super;
  Val x856 = x4[51]._super._super;
  Val x857 = x4[52]._super._super;
  Val x858 = x4[53]._super._super;
  Val x859 = x4[54]._super._super;
  Val x860 = x4[55]._super._super;
  Val x861 = x4[56]._super._super;
  Val x862 = x4[57]._super._super;
  Val x863 = x4[58]._super._super;
  Val x864 = x4[59]._super._super;
  Val x865 = x4[60]._super._super;
  Val x866 = x4[61]._super._super;
  Val x867 = x4[62]._super._super;
  Val x868 = x4[63]._super._super;
  Val x869 = x4[64]._super._super;
  Val x870 = x4[65]._super._super;
  Val x871 = x4[66]._super._super;
  Val x872 = x4[67]._super._super;
  Val x873 = x4[68]._super._super;
  Val x874 = x4[69]._super._super;
  Val x875 = x4[70]._super._super;
  Val x876 = x4[71]._super._super;
  Val x877 = x4[72]._super._super;
  Val x878 = x4[73]._super._super;
  Val x879 = x4[74]._super._super;
  Val x880 = x4[75]._super._super;
  Val x881 = x4[76]._super._super;
  Val x882 = x4[77]._super._super;
  Val x883 = x4[78]._super._super;
  Val x884 = x4[79]._super._super;
  Val x885 = x4[80]._super._super;
  Val x886 = x4[81]._super._super;
  Val x887 = x4[82]._super._super;
  Val x888 = x4[83]._super._super;
  Val x889 = x4[84]._super._super;
  Val x890 = x4[85]._super._super;
  Val x891 = x4[86]._super._super;
  Val x892 = x4[87]._super._super;
  Val x893 = x4[88]._super._super;
  Val x894 = x4[89]._super._super;
  Val x895 = x4[90]._super._super;
  Val x896 = x4[91]._super._super;
  Val x897 = x4[92]._super._super;
  Val x898 = x4[93]._super._super;
  Val x899 = x4[94]._super._super;
  Val x900 = x4[95]._super._super;
  Val x901 = x4[96]._super._super;
  Val x902 = x4[97]._super._super;
  Val x903 = x4[98]._super._super;
  Val x904 = x4[99]._super._super;
  Val x905 = x4[100]._super._super;
  Val x906 = x4[101]._super._super;
  Val x907 = x4[102]._super._super;
  Val x908 = x4[103]._super._super;
  Val x909 = x4[104]._super._super;
  Val x910 = x4[105]._super._super;
  Val x911 = x4[106]._super._super;
  Val x912 = x4[107]._super._super;
  Val x913 = x4[108]._super._super;
  Val x914 = x4[109]._super._super;
  Val x915 = x4[110]._super._super;
  Val x916 = x4[111]._super._super;
  Val x917 = x4[112]._super._super;
  Val x918 = x4[113]._super._super;
  Val x919 = x4[114]._super._super;
  Val x920 = x4[115]._super._super;
  Val x921 = x4[116]._super._super;
  Val x922 = x4[117]._super._super;
  Val x923 = x4[118]._super._super;
  Val x924 = x4[119]._super._super;
  Val x925 = x4[120]._super._super;
  Val x926 = x4[121]._super._super;
  Val x927 = x4[122]._super._super;
  Val x928 = x4[123]._super._super;
  Val x929 = x4[124]._super._super;
  Val x930 = x4[125]._super._super;
  Val x931 = x4[126]._super._super;
  Val x932 = x4[127]._super._super;
  Val x933 = x4[128]._super._super;
  Val x934 = x4[129]._super._super;
  Val x935 = x4[130]._super._super;
  Val x936 = x4[131]._super._super;
  Val x937 = x4[132]._super._super;
  Val x938 = x4[133]._super._super;
  Val x939 = x4[134]._super._super;
  Val x940 = x4[135]._super._super;
  Val x941 = x4[136]._super._super;
  Val x942 = x4[137]._super._super;
  Val x943 = x4[138]._super._super;
  Val x944 = x4[139]._super._super;
  Val x945 = x4[140]._super._super;
  Val x946 = x4[141]._super._super;
  Val x947 = x4[142]._super._super;
  Val x948 = x4[143]._super._super;
  Val x949 = x4[144]._super._super;
  Val x950 = x4[145]._super._super;
  Val x951 = x4[146]._super._super;
  Val x952 = x4[147]._super._super;
  Val x953 = x4[148]._super._super;
  Val x954 = x4[149]._super._super;
  Val x955 = x4[150]._super._super;
  Val x956 = x4[151]._super._super;
  Val x957 = x4[152]._super._super;
  Val x958 = x4[153]._super._super;
  Val x959 = x4[154]._super._super;
  Val x960 = x4[155]._super._super;
  Val x961 = x4[156]._super._super;
  Val x962 = x4[157]._super._super;
  Val x963 = x4[158]._super._super;
  Val x964 = x4[159]._super._super;
  Val x965 = x4[160]._super._super;
  Val x966 = x4[161]._super._super;
  Val x967 = x4[162]._super._super;
  Val x968 = x4[163]._super._super;
  Val x969 = x4[164]._super._super;
  Val x970 = x4[165]._super._super;
  Val x971 = x4[166]._super._super;
  Val x972 = x4[167]._super._super;
  Val x973 = x4[168]._super._super;
  Val x974 = x4[169]._super._super;
  Val x975 = x4[170]._super._super;
  Val x976 = x4[171]._super._super;
  Val x977 = x4[172]._super._super;
  Val x978 = x4[173]._super._super;
  Val x979 = x4[174]._super._super;
  Val x980 = x4[175]._super._super;
  Val x981 = x4[176]._super._super;
  Val x982 = x4[177]._super._super;
  Val x983 = x4[178]._super._super;
  Val x984 = x4[179]._super._super;
  Val x985 = x4[180]._super._super;
  Val x986 = x4[181]._super._super;
  Val x987 = x4[182]._super._super;
  Val x988 = x4[183]._super._super;
  Val x989 = x4[184]._super._super;
  Val x990 = x4[185]._super._super;
  Val x991 = x4[186]._super._super;
  Val x992 = x4[187]._super._super;
  Val x993 = x4[188]._super._super;
  Val x994 = x4[189]._super._super;
  Val x995 = x4[190]._super._super;
  Val x996 = x4[191]._super._super;
  Val x997 = x4[192]._super._super;
  Val x998 = x4[193]._super._super;
  Val x999 = x4[194]._super._super;
  Val x1000 = x4[195]._super._super;
  Val x1001 = x4[196]._super._super;
  Val x1002 = x4[197]._super._super;
  Val x1003 = x4[198]._super._super;
  Val x1004 = x4[199]._super._super;
  Val x1005 = x4[200]._super._super;
  Val x1006 = x4[201]._super._super;
  Val x1007 = x4[202]._super._super;
  Val x1008 = x4[203]._super._super;
  Val x1009 = x4[204]._super._super;
  Val x1010 = x4[205]._super._super;
  Val x1011 = x4[206]._super._super;
  Val x1012 = x4[207]._super._super;
  Val x1013 = x4[208]._super._super;
  Val x1014 = x4[209]._super._super;
  Val x1015 = x4[210]._super._super;
  Val x1016 = x4[211]._super._super;
  Val x1017 = x4[212]._super._super;
  Val x1018 = x4[213]._super._super;
  Val x1019 = x4[214]._super._super;
  Val x1020 = x4[215]._super._super;
  Val x1021 = x4[216]._super._super;
  Val x1022 = x4[217]._super._super;
  Val x1023 = x4[218]._super._super;
  Val x1024 = x4[219]._super._super;
  Val x1025 = x4[220]._super._super;
  Val x1026 = x4[221]._super._super;
  Val x1027 = x4[222]._super._super;
  Val x1028 = x4[223]._super._super;
  Val x1029 = x4[224]._super._super;
  Val x1030 = x4[225]._super._super;
  Val x1031 = x4[226]._super._super;
  Val x1032 = x4[227]._super._super;
  Val x1033 = x4[228]._super._super;
  Val x1034 = x4[229]._super._super;
  Val x1035 = x4[230]._super._super;
  Val x1036 = x4[231]._super._super;
  Val x1037 = x4[232]._super._super;
  Val x1038 = x4[233]._super._super;
  Val x1039 = x4[234]._super._super;
  Val x1040 = x4[235]._super._super;
  Val x1041 = x4[236]._super._super;
  Val x1042 = x4[237]._super._super;
  Val x1043 = x4[238]._super._super;
  Val x1044 = x4[239]._super._super;
  Val x1045 = x4[240]._super._super;
  Val x1046 = x4[241]._super._super;
  Val x1047 = x4[242]._super._super;
  Val x1048 = x4[243]._super._super;
  Val x1049 = x4[244]._super._super;
  Val x1050 = x4[245]._super._super;
  Val x1051 = x4[246]._super._super;
  Val x1052 = x4[247]._super._super;
  Val x1053 = x4[248]._super._super;
  Val x1054 = x4[249]._super._super;
  Val x1055 = x4[250]._super._super;
  Val x1056 = x4[251]._super._super;
  Val x1057 = x4[252]._super._super;
  Val x1058 = x4[253]._super._super;
  Val x1059 = x4[254]._super._super;
  Val x1060 = x4[255]._super._super;
  Val x1061 = x4[256]._super._super;
  Val x1062 = x4[257]._super._super;
  Val x1063 = x4[258]._super._super;
  Val x1064 = x4[259]._super._super;
  Val x1065 = x4[260]._super._super;
  Val x1066 = x4[261]._super._super;
  Val x1067 = x4[262]._super._super;
  Val x1068 = x4[263]._super._super;
  Val x1069 = x4[264]._super._super;
  Val x1070 = x4[265]._super._super;
  Val x1071 = x4[266]._super._super;
  Val x1072 = x4[267]._super._super;
  Val x1073 = x4[268]._super._super;
  Val x1074 = x4[269]._super._super;
  Val x1075 = x4[270]._super._super;
  Val x1076 = x4[271]._super._super;
  Val x1077 = x4[272]._super._super;
  Val x1078 = x4[273]._super._super;
  Val x1079 = x4[274]._super._super;
  Val x1080 = x4[275]._super._super;
  Val x1081 = x4[276]._super._super;
  Val x1082 = x4[277]._super._super;
  Val x1083 = x4[278]._super._super;
  Val x1084 = x4[279]._super._super;
  Val x1085 = x4[280]._super._super;
  Val x1086 = x4[281]._super._super;
  Val x1087 = x4[282]._super._super;
  Val x1088 = x4[283]._super._super;
  Val x1089 = x4[284]._super._super;
  Val x1090 = x4[285]._super._super;
  Val x1091 = x4[286]._super._super;
  Val x1092 = x4[287]._super._super;
  Val x1093 = x4[288]._super._super;
  Val x1094 = x4[289]._super._super;
  Val x1095 = x4[290]._super._super;
  Val x1096 = x4[291]._super._super;
  Val x1097 = x4[292]._super._super;
  Val x1098 = x4[293]._super._super;
  Val x1099 = x4[294]._super._super;
  Val x1100 = x4[295]._super._super;
  Val x1101 = x4[296]._super._super;
  Val x1102 = x4[297]._super._super;
  Val x1103 = x4[298]._super._super;
  Val x1104 = x4[299]._super._super;
  Val x1105 = x4[300]._super._super;
  Val x1106 = x4[301]._super._super;
  Val x1107 = x4[302]._super._super;
  Val x1108 = x4[303]._super._super;
  Val x1109 = x4[304]._super._super;
  Val x1110 = x4[305]._super._super;
  Val x1111 = x4[306]._super._super;
  Val x1112 = x4[307]._super._super;
  Val x1113 = x4[308]._super._super;
  Val x1114 = x4[309]._super._super;
  Val x1115 = x4[310]._super._super;
  Val x1116 = x4[311]._super._super;
  Val x1117 = x4[312]._super._super;
  Val x1118 = x4[313]._super._super;
  Val x1119 = x4[314]._super._super;
  Val x1120 = x4[315]._super._super;
  Val x1121 = x4[316]._super._super;
  Val x1122 = x4[317]._super._super;
  Val x1123 = x4[318]._super._super;
  Val x1124 = x4[319]._super._super;
  Val x1125 = x4[320]._super._super;
  Val x1126 = x4[321]._super._super;
  Val x1127 = x4[322]._super._super;
  Val x1128 = x4[323]._super._super;
  Val x1129 = x4[324]._super._super;
  Val x1130 = x4[325]._super._super;
  Val x1131 = x4[326]._super._super;
  Val x1132 = x4[327]._super._super;
  Val x1133 = x4[328]._super._super;
  Val x1134 = x4[329]._super._super;
  Val x1135 = x4[330]._super._super;
  Val x1136 = x4[331]._super._super;
  Val x1137 = x4[332]._super._super;
  Val x1138 = x4[333]._super._super;
  Val x1139 = x4[334]._super._super;
  Val x1140 = x4[335]._super._super;
  Val x1141 = x4[336]._super._super;
  Val x1142 = x4[337]._super._super;
  Val x1143 = x4[338]._super._super;
  Val x1144 = x4[339]._super._super;
  Val x1145 = x4[340]._super._super;
  Val x1146 = x4[341]._super._super;
  Val x1147 = x4[342]._super._super;
  Val x1148 = x4[343]._super._super;
  Val x1149 = x4[344]._super._super;
  Val x1150 = x4[345]._super._super;
  Val x1151 = x4[346]._super._super;
  Val x1152 = x4[347]._super._super;
  Val x1153 = x4[348]._super._super;
  Val x1154 = x4[349]._super._super;
  Val x1155 = x4[350]._super._super;
  Val x1156 = x4[351]._super._super;
  Val x1157 = x4[352]._super._super;
  Val x1158 = x4[353]._super._super;
  Val x1159 = x4[354]._super._super;
  Val x1160 = x4[355]._super._super;
  Val x1161 = x4[356]._super._super;
  Val x1162 = x4[357]._super._super;
  Val x1163 = x4[358]._super._super;
  Val x1164 = x4[359]._super._super;
  Val x1165 = x4[360]._super._super;
  Val x1166 = x4[361]._super._super;
  Val x1167 = x4[362]._super._super;
  Val x1168 = x4[363]._super._super;
  Val x1169 = x4[364]._super._super;
  Val x1170 = x4[365]._super._super;
  Val x1171 = x4[366]._super._super;
  Val x1172 = x4[367]._super._super;
  Val x1173 = x4[368]._super._super;
  Val x1174 = x4[369]._super._super;
  Val x1175 = x4[370]._super._super;
  Val x1176 = x4[371]._super._super;
  Val x1177 = x4[372]._super._super;
  Val x1178 = x4[373]._super._super;
  Val x1179 = x4[374]._super._super;
  Val x1180 = x4[375]._super._super;
  Val x1181 = x4[376]._super._super;
  Val x1182 = x4[377]._super._super;
  Val x1183 = x4[378]._super._super;
  Val x1184 = x4[379]._super._super;
  Val x1185 = x4[380]._super._super;
  Val x1186 = x4[381]._super._super;
  Val x1187 = x4[382]._super._super;
  Val x1188 = x4[383]._super._super;
  Val x1189 = x4[384]._super._super;
  Val x1190 = x4[385]._super._super;
  Val x1191 = x4[386]._super._super;
  Val x1192 = x4[387]._super._super;
  Val x1193 = x4[388]._super._super;
  Val x1194 = x4[389]._super._super;
  Val x1195 = x4[390]._super._super;
  Val x1196 = x4[391]._super._super;
  Val x1197 = x4[392]._super._super;
  Val x1198 = x4[393]._super._super;
  Val x1199 = x4[394]._super._super;
  Val x1200 = x4[395]._super._super;
  Val x1201 = x4[396]._super._super;
  Val x1202 = x4[397]._super._super;
  Val x1203 = x4[398]._super._super;
  Val x1204 = x4[399]._super._super;
  Val x1205 = x4[400]._super._super;
  Val x1206 = x4[401]._super._super;
  Val x1207 = x4[402]._super._super;
  Val x1208 = x4[403]._super._super;
  Val x1209 = x4[404]._super._super;
  Val x1210 = x4[405]._super._super;
  Val x1211 = x4[406]._super._super;
  Val x1212 = x4[407]._super._super;
  Val x1213 = x4[408]._super._super;
  Val x1214 = x4[409]._super._super;
  Val x1215 = x4[410]._super._super;
  Val x1216 = x4[411]._super._super;
  Val x1217 = x4[412]._super._super;
  Val x1218 = x4[413]._super._super;
  Val x1219 = x4[414]._super._super;
  Val x1220 = x4[415]._super._super;
  Val x1221 = x4[416]._super._super;
  Val x1222 = x4[417]._super._super;
  Val x1223 = x4[418]._super._super;
  Val x1224 = x4[419]._super._super;
  Val x1225 = x4[420]._super._super;
  Val x1226 = x4[421]._super._super;
  Val x1227 = x4[422]._super._super;
  Val x1228 = x4[423]._super._super;
  Val x1229 = x4[424]._super._super;
  Val x1230 = x4[425]._super._super;
  Val x1231 = x4[426]._super._super;
  Val x1232 = x4[427]._super._super;
  Val x1233 = x4[428]._super._super;
  Val x1234 = x4[429]._super._super;
  Val x1235 = x4[430]._super._super;
  Val x1236 = x4[431]._super._super;
  Val x1237 = x4[432]._super._super;
  Val x1238 = x4[433]._super._super;
  Val x1239 = x4[434]._super._super;
  Val x1240 = x4[435]._super._super;
  Val x1241 = x4[436]._super._super;
  Val x1242 = x4[437]._super._super;
  Val x1243 = x4[438]._super._super;
  Val x1244 = x4[439]._super._super;
  Val x1245 = x4[440]._super._super;
  Val x1246 = x4[441]._super._super;
  Val x1247 = x4[442]._super._super;
  Val x1248 = x4[443]._super._super;
  Val x1249 = x4[444]._super._super;
  Val x1250 = x4[445]._super._super;
  Val x1251 = x4[446]._super._super;
  Val x1252 = x4[447]._super._super;
  Val x1253 = x4[448]._super._super;
  Val x1254 = x4[449]._super._super;
  Val x1255 = x4[450]._super._super;
  Val x1256 = x4[451]._super._super;
  Val x1257 = x4[452]._super._super;
  Val x1258 = x4[453]._super._super;
  Val x1259 = x4[454]._super._super;
  Val x1260 = x4[455]._super._super;
  Val x1261 = x4[456]._super._super;
  Val x1262 = x4[457]._super._super;
  Val x1263 = x4[458]._super._super;
  Val x1264 = x4[459]._super._super;
  Val x1265 = x4[460]._super._super;
  Val x1266 = x4[461]._super._super;
  Val x1267 = x4[462]._super._super;
  Val x1268 = x4[463]._super._super;
  Val x1269 = x4[464]._super._super;
  Val x1270 = x4[465]._super._super;
  Val x1271 = x4[466]._super._super;
  Val x1272 = x4[467]._super._super;
  Val x1273 = x4[468]._super._super;
  Val x1274 = x4[469]._super._super;
  Val x1275 = x4[470]._super._super;
  Val x1276 = x4[471]._super._super;
  Val x1277 = x4[472]._super._super;
  Val x1278 = x4[473]._super._super;
  Val x1279 = x4[474]._super._super;
  Val x1280 = x4[475]._super._super;
  Val x1281 = x4[476]._super._super;
  Val x1282 = x4[477]._super._super;
  Val x1283 = x4[478]._super._super;
  Val x1284 = x4[479]._super._super;
  Val x1285 = x4[480]._super._super;
  Val x1286 = x4[481]._super._super;
  Val x1287 = x4[482]._super._super;
  Val x1288 = x4[483]._super._super;
  Val x1289 = x4[484]._super._super;
  Val x1290 = x4[485]._super._super;
  Val x1291 = x4[486]._super._super;
  Val x1292 = x4[487]._super._super;
  Val x1293 = x4[488]._super._super;
  Val x1294 = x4[489]._super._super;
  Val x1295 = x4[490]._super._super;
  Val x1296 = x4[491]._super._super;
  Val x1297 = x4[492]._super._super;
  Val x1298 = x4[493]._super._super;
  Val x1299 = x4[494]._super._super;
  Val x1300 = x4[495]._super._super;
  Val x1301 = x4[496]._super._super;
  Val x1302 = x4[497]._super._super;
  Val x1303 = x4[498]._super._super;
  Val x1304 = x4[499]._super._super;
  Val x1305 = x4[500]._super._super;
  Val x1306 = x4[501]._super._super;
  Val x1307 = x4[502]._super._super;
  Val x1308 = x4[503]._super._super;
  Val x1309 = x4[504]._super._super;
  Val x1310 = x4[505]._super._super;
  Val x1311 = x4[506]._super._super;
  Val x1312 = x4[507]._super._super;
  Val x1313 = x4[508]._super._super;
  Val x1314 = x4[509]._super._super;
  Val x1315 = x4[510]._super._super;
  Val x1316 = x4[511]._super._super;
  Val x1317 = x4[512]._super._super;
  Val x1318 = x4[513]._super._super;
  Val x1319 = x4[514]._super._super;
  Val x1320 = x4[515]._super._super;
  Val x1321 = x4[516]._super._super;
  Val x1322 = x4[517]._super._super;
  Val x1323 = x4[518]._super._super;
  Val x1324 = x4[519]._super._super;
  Val x1325 = x4[520]._super._super;
  Val x1326 = x4[521]._super._super;
  Val x1327 = x4[522]._super._super;
  Val x1328 = x4[523]._super._super;
  Val x1329 = x4[524]._super._super;
  Val x1330 = x4[525]._super._super;
  Val x1331 = x4[526]._super._super;
  Val x1332 = x4[527]._super._super;
  Val x1333 = x4[528]._super._super;
  Val x1334 = x4[529]._super._super;
  Val x1335 = x4[530]._super._super;
  Val x1336 = x4[531]._super._super;
  Val x1337 = x4[532]._super._super;
  Val x1338 = x4[533]._super._super;
  Val x1339 = x4[534]._super._super;
  Val x1340 = x4[535]._super._super;
  Val x1341 = x4[536]._super._super;
  Val x1342 = x4[537]._super._super;
  Val x1343 = x4[538]._super._super;
  Val x1344 = x4[539]._super._super;
  Val x1345 = x4[540]._super._super;
  Val x1346 = x4[541]._super._super;
  Val x1347 = x4[542]._super._super;
  Val x1348 = x4[543]._super._super;
  Val x1349 = x4[544]._super._super;
  Val x1350 = x4[545]._super._super;
  Val x1351 = x4[546]._super._super;
  Val x1352 = x4[547]._super._super;
  Val x1353 = x4[548]._super._super;
  Val x1354 = x4[549]._super._super;
  Val x1355 = x4[550]._super._super;
  Val x1356 = x4[551]._super._super;
  Val x1357 = x4[552]._super._super;
  Val x1358 = x4[553]._super._super;
  Val x1359 = x4[554]._super._super;
  Val x1360 = x4[555]._super._super;
  Val x1361 = x4[556]._super._super;
  Val x1362 = x4[557]._super._super;
  Val x1363 = x4[558]._super._super;
  Val x1364 = x4[559]._super._super;
  Val x1365 = x4[560]._super._super;
  Val x1366 = x4[561]._super._super;
  Val x1367 = x4[562]._super._super;
  Val x1368 = x4[563]._super._super;
  Val x1369 = x4[564]._super._super;
  Val x1370 = x4[565]._super._super;
  Val x1371 = x4[566]._super._super;
  Val x1372 = x4[567]._super._super;
  Val x1373 = x4[568]._super._super;
  Val x1374 = x4[569]._super._super;
  Val x1375 = x4[570]._super._super;
  Val x1376 = x4[571]._super._super;
  Val x1377 = x4[572]._super._super;
  Val x1378 = x4[573]._super._super;
  Val x1379 = x4[574]._super._super;
  Val x1380 = x4[575]._super._super;
  Val x1381 = x4[576]._super._super;
  Val x1382 = x4[577]._super._super;
  Val x1383 = x4[578]._super._super;
  Val x1384 = x4[579]._super._super;
  Val x1385 = x4[580]._super._super;
  Val x1386 = x4[581]._super._super;
  Val x1387 = x4[582]._super._super;
  Val x1388 = x4[583]._super._super;
  Val x1389 = x4[584]._super._super;
  Val x1390 = x4[585]._super._super;
  Val x1391 = x4[586]._super._super;
  Val x1392 = x4[587]._super._super;
  Val x1393 = x4[588]._super._super;
  Val x1394 = x4[589]._super._super;
  Val x1395 = x4[590]._super._super;
  Val x1396 = x4[591]._super._super;
  Val x1397 = x4[592]._super._super;
  Val x1398 = x4[593]._super._super;
  Val x1399 = x4[594]._super._super;
  Val x1400 = x4[595]._super._super;
  Val x1401 = x4[596]._super._super;
  Val x1402 = x4[597]._super._super;
  Val x1403 = x4[598]._super._super;
  Val x1404 = x4[599]._super._super;
  Val x1405 = x4[600]._super._super;
  Val x1406 = x4[601]._super._super;
  Val x1407 = x4[602]._super._super;
  Val x1408 = x4[603]._super._super;
  Val x1409 = x4[604]._super._super;
  Val x1410 = x4[605]._super._super;
  Val x1411 = x4[606]._super._super;
  Val x1412 = x4[607]._super._super;
  Val x1413 = x4[608]._super._super;
  Val x1414 = x4[609]._super._super;
  Val x1415 = x4[610]._super._super;
  Val x1416 = x4[611]._super._super;
  Val x1417 = x4[612]._super._super;
  Val x1418 = x4[613]._super._super;
  Val x1419 = x4[614]._super._super;
  Val x1420 = x4[615]._super._super;
  Val x1421 = x4[616]._super._super;
  Val x1422 = x4[617]._super._super;
  Val x1423 = x4[618]._super._super;
  Val x1424 = x4[619]._super._super;
  Val x1425 = x4[620]._super._super;
  Val x1426 = x4[621]._super._super;
  Val x1427 = x4[622]._super._super;
  Val x1428 = x4[623]._super._super;
  Val x1429 = x4[624]._super._super;
  Val x1430 = x4[625]._super._super;
  Val x1431 = x4[626]._super._super;
  Val x1432 = x4[627]._super._super;
  Val x1433 = x4[628]._super._super;
  Val x1434 = x4[629]._super._super;
  Val x1435 = x4[630]._super._super;
  Val x1436 = x4[631]._super._super;
  Val x1437 = x4[632]._super._super;
  Val x1438 = x4[633]._super._super;
  Val x1439 = x4[634]._super._super;
  Val x1440 = x4[635]._super._super;
  Val x1441 = x4[636]._super._super;
  Val x1442 = x4[637]._super._super;
  Val x1443 = x4[638]._super._super;
  Val x1444 = x4[639]._super._super;
  Val x1445 = x4[640]._super._super;
  Val x1446 = x4[641]._super._super;
  Val x1447 = x4[642]._super._super;
  Val x1448 = x4[643]._super._super;
  Val x1449 = x4[644]._super._super;
  Val x1450 = x4[645]._super._super;
  Val x1451 = x4[646]._super._super;
  Val x1452 = x4[647]._super._super;
  Val x1453 = x4[648]._super._super;
  Val x1454 = x4[649]._super._super;
  Val x1455 = x4[650]._super._super;
  Val x1456 = x4[651]._super._super;
  Val x1457 = x4[652]._super._super;
  Val x1458 = x4[653]._super._super;
  Val x1459 = x4[654]._super._super;
  Val x1460 = x4[655]._super._super;
  Val x1461 = x4[656]._super._super;
  Val x1462 = x4[657]._super._super;
  Val x1463 = x4[658]._super._super;
  Val x1464 = x4[659]._super._super;
  Val x1465 = x4[660]._super._super;
  Val x1466 = x4[661]._super._super;
  Val x1467 = x4[662]._super._super;
  Val x1468 = x4[663]._super._super;
  Val x1469 = x4[664]._super._super;
  Val x1470 = x4[665]._super._super;
  Val x1471 = x4[666]._super._super;
  Val x1472 = x4[667]._super._super;
  Val x1473 = x4[668]._super._super;
  Val x1474 = x4[669]._super._super;
  Val x1475 = x4[670]._super._super;
  Val x1476 = x4[671]._super._super;
  Val x1477 = x4[672]._super._super;
  Val x1478 = x4[673]._super._super;
  Val x1479 = x4[674]._super._super;
  Val x1480 = x4[675]._super._super;
  Val x1481 = x4[676]._super._super;
  Val x1482 = x4[677]._super._super;
  Val x1483 = x4[678]._super._super;
  Val x1484 = x4[679]._super._super;
  Val x1485 = x4[680]._super._super;
  Val x1486 = x4[681]._super._super;
  Val x1487 = x4[682]._super._super;
  Val x1488 = x4[683]._super._super;
  Val x1489 = x4[684]._super._super;
  Val x1490 = x4[685]._super._super;
  Val x1491 = x4[686]._super._super;
  Val x1492 = x4[687]._super._super;
  Val x1493 = x4[688]._super._super;
  Val x1494 = x4[689]._super._super;
  Val x1495 = x4[690]._super._super;
  Val x1496 = x4[691]._super._super;
  Val x1497 = x4[692]._super._super;
  Val x1498 = x4[693]._super._super;
  Val x1499 = x4[694]._super._super;
  Val x1500 = x4[695]._super._super;
  Val x1501 = x4[696]._super._super;
  Val x1502 = x4[697]._super._super;
  Val x1503 = x4[698]._super._super;
  Val x1504 = x4[699]._super._super;
  Val x1505 = x4[700]._super._super;
  Val x1506 = x4[701]._super._super;
  Val x1507 = x4[702]._super._super;
  Val x1508 = x4[703]._super._super;
  Val x1509 = x4[704]._super._super;
  Val x1510 = x4[705]._super._super;
  Val x1511 = x4[706]._super._super;
  Val x1512 = x4[707]._super._super;
  Val x1513 = x4[708]._super._super;
  Val x1514 = x4[709]._super._super;
  Val x1515 = x4[710]._super._super;
  Val x1516 = x4[711]._super._super;
  Val x1517 = x4[712]._super._super;
  Val x1518 = x4[713]._super._super;
  Val x1519 = x4[714]._super._super;
  Val x1520 = x4[715]._super._super;
  Val x1521 = x4[716]._super._super;
  Val x1522 = x4[717]._super._super;
  Val x1523 = x4[718]._super._super;
  Val x1524 = x4[719]._super._super;
  Val x1525 = x4[720]._super._super;
  Val x1526 = x4[721]._super._super;
  Val x1527 = x4[722]._super._super;
  Val x1528 = x4[723]._super._super;
  Val x1529 = x4[724]._super._super;
  Val x1530 = x4[725]._super._super;
  Val x1531 = x4[726]._super._super;
  Val x1532 = x4[727]._super._super;
  Val x1533 = x4[728]._super._super;
  Val x1534 = x4[729]._super._super;
  Val x1535 = x4[730]._super._super;
  Val x1536 = x4[731]._super._super;
  Val x1537 = x4[732]._super._super;
  Val x1538 = x4[733]._super._super;
  Val x1539 = x4[734]._super._super;
  Val x1540 = x4[735]._super._super;
  Val x1541 = x4[736]._super._super;
  Val x1542 = x4[737]._super._super;
  Val x1543 = x4[738]._super._super;
  Val x1544 = x4[739]._super._super;
  Val x1545 = x4[740]._super._super;
  Val x1546 = x4[741]._super._super;
  Val x1547 = x4[742]._super._super;
  Val x1548 = x4[743]._super._super;
  Val x1549 = x4[744]._super._super;
  Val x1550 = x4[745]._super._super;
  Val x1551 = x4[746]._super._super;
  Val x1552 = x4[747]._super._super;
  Val x1553 = x4[748]._super._super;
  Val x1554 = x4[749]._super._super;
  Val x1555 = x4[750]._super._super;
  Val x1556 = x4[751]._super._super;
  Val x1557 = x4[752]._super._super;
  Val x1558 = x4[753]._super._super;
  Val x1559 = x4[754]._super._super;
  Val x1560 = x4[755]._super._super;
  Val x1561 = x4[756]._super._super;
  Val x1562 = x4[757]._super._super;
  Val x1563 = x4[758]._super._super;
  Val x1564 = x4[759]._super._super;
  Val x1565 = x4[760]._super._super;
  Val x1566 = x4[761]._super._super;
  Val x1567 = x4[762]._super._super;
  Val x1568 = x4[763]._super._super;
  Val x1569 = x4[764]._super._super;
  Val x1570 = x4[765]._super._super;
  Val x1571 = x4[766]._super._super;
  Val x1572 = x4[767]._super._super;
  Val x1573 = x4[768]._super._super;
  Val x1574 = x4[769]._super._super;
  Val x1575 = x4[770]._super._super;
  Val x1576 = x4[771]._super._super;
  Val x1577 = x4[772]._super._super;
  Val x1578 = x4[773]._super._super;
  Val x1579 = x4[774]._super._super;
  Val x1580 = x4[775]._super._super;
  Val x1581 = x4[776]._super._super;
  Val x1582 = x4[777]._super._super;
  Val x1583 = x4[778]._super._super;
  Val x1584 = x4[779]._super._super;
  Val x1585 = x4[780]._super._super;
  Val x1586 = x4[781]._super._super;
  Val x1587 = x4[782]._super._super;
  Val x1588 = x4[783]._super._super;
  Val x1589 = x4[784]._super._super;
  Val x1590 = x4[785]._super._super;
  Val x1591 = x4[786]._super._super;
  Val x1592 = x4[787]._super._super;
  Val x1593 = x4[788]._super._super;
  Val x1594 = x4[789]._super._super;
  Val x1595 = x4[790]._super._super;
  Val x1596 = x4[791]._super._super;
  Val x1597 = x4[792]._super._super;
  Val x1598 = x4[793]._super._super;
  Val x1599 = x4[794]._super._super;
  Val x1600 = x4[795]._super._super;
  Val x1601 = x4[796]._super._super;
  Val x1602 = x4[797]._super._super;
  Val x1603 = x4[798]._super._super;
  Val x1604 = x4[799]._super._super;
  Val64Array5Array5Array x1605 = Val64Array5Array5Array{
      Val64Array5Array{
          Val64Array{x5,   x6,   x7,   x8,   x9,   x10,  x11,  x12,  x13,  x14,  x15,  x16,  x17,
                     x18,  x19,  x20,  x21,  x22,  x23,  x24,  x25,  x26,  x27,  x28,  x29,  x30,
                     x31,  x32,  x33,  x34,  x35,  x36,  x805, x806, x807, x808, x809, x810, x811,
                     x812, x813, x814, x815, x816, x817, x818, x819, x820, x821, x822, x823, x824,
                     x825, x826, x827, x828, x829, x830, x831, x832, x833, x834, x835, x836},
          Val64Array{x37,  x38,  x39,  x40,  x41,  x42,  x43,  x44,  x45,  x46,  x47,  x48,  x49,
                     x50,  x51,  x52,  x53,  x54,  x55,  x56,  x57,  x58,  x59,  x60,  x61,  x62,
                     x63,  x64,  x65,  x66,  x67,  x68,  x837, x838, x839, x840, x841, x842, x843,
                     x844, x845, x846, x847, x848, x849, x850, x851, x852, x853, x854, x855, x856,
                     x857, x858, x859, x860, x861, x862, x863, x864, x865, x866, x867, x868},
          Val64Array{x69,  x70,  x71,  x72,  x73,  x74,  x75,  x76,  x77,  x78,  x79,  x80,  x81,
                     x82,  x83,  x84,  x85,  x86,  x87,  x88,  x89,  x90,  x91,  x92,  x93,  x94,
                     x95,  x96,  x97,  x98,  x99,  x100, x869, x870, x871, x872, x873, x874, x875,
                     x876, x877, x878, x879, x880, x881, x882, x883, x884, x885, x886, x887, x888,
                     x889, x890, x891, x892, x893, x894, x895, x896, x897, x898, x899, x900},
          Val64Array{x101, x102, x103, x104, x105, x106, x107, x108, x109, x110, x111, x112, x113,
                     x114, x115, x116, x117, x118, x119, x120, x121, x122, x123, x124, x125, x126,
                     x127, x128, x129, x130, x131, x132, x901, x902, x903, x904, x905, x906, x907,
                     x908, x909, x910, x911, x912, x913, x914, x915, x916, x917, x918, x919, x920,
                     x921, x922, x923, x924, x925, x926, x927, x928, x929, x930, x931, x932},
          Val64Array{x133, x134, x135, x136, x137, x138, x139, x140, x141, x142, x143, x144, x145,
                     x146, x147, x148, x149, x150, x151, x152, x153, x154, x155, x156, x157, x158,
                     x159, x160, x161, x162, x163, x164, x933, x934, x935, x936, x937, x938, x939,
                     x940, x941, x942, x943, x944, x945, x946, x947, x948, x949, x950, x951, x952,
                     x953, x954, x955, x956, x957, x958, x959, x960, x961, x962, x963, x964}},
      Val64Array5Array{
          Val64Array{x165, x166, x167, x168, x169, x170, x171, x172, x173, x174, x175, x176, x177,
                     x178, x179, x180, x181, x182, x183, x184, x185, x186, x187, x188, x189, x190,
                     x191, x192, x193, x194, x195, x196, x965, x966, x967, x968, x969, x970, x971,
                     x972, x973, x974, x975, x976, x977, x978, x979, x980, x981, x982, x983, x984,
                     x985, x986, x987, x988, x989, x990, x991, x992, x993, x994, x995, x996},
          Val64Array{x197,  x198,  x199,  x200,  x201,  x202,  x203,  x204,  x205,  x206,  x207,
                     x208,  x209,  x210,  x211,  x212,  x213,  x214,  x215,  x216,  x217,  x218,
                     x219,  x220,  x221,  x222,  x223,  x224,  x225,  x226,  x227,  x228,  x997,
                     x998,  x999,  x1000, x1001, x1002, x1003, x1004, x1005, x1006, x1007, x1008,
                     x1009, x1010, x1011, x1012, x1013, x1014, x1015, x1016, x1017, x1018, x1019,
                     x1020, x1021, x1022, x1023, x1024, x1025, x1026, x1027, x1028},
          Val64Array{x229,  x230,  x231,  x232,  x233,  x234,  x235,  x236,  x237,  x238,  x239,
                     x240,  x241,  x242,  x243,  x244,  x245,  x246,  x247,  x248,  x249,  x250,
                     x251,  x252,  x253,  x254,  x255,  x256,  x257,  x258,  x259,  x260,  x1029,
                     x1030, x1031, x1032, x1033, x1034, x1035, x1036, x1037, x1038, x1039, x1040,
                     x1041, x1042, x1043, x1044, x1045, x1046, x1047, x1048, x1049, x1050, x1051,
                     x1052, x1053, x1054, x1055, x1056, x1057, x1058, x1059, x1060},
          Val64Array{x261,  x262,  x263,  x264,  x265,  x266,  x267,  x268,  x269,  x270,  x271,
                     x272,  x273,  x274,  x275,  x276,  x277,  x278,  x279,  x280,  x281,  x282,
                     x283,  x284,  x285,  x286,  x287,  x288,  x289,  x290,  x291,  x292,  x1061,
                     x1062, x1063, x1064, x1065, x1066, x1067, x1068, x1069, x1070, x1071, x1072,
                     x1073, x1074, x1075, x1076, x1077, x1078, x1079, x1080, x1081, x1082, x1083,
                     x1084, x1085, x1086, x1087, x1088, x1089, x1090, x1091, x1092},
          Val64Array{x293,  x294,  x295,  x296,  x297,  x298,  x299,  x300,  x301,  x302,  x303,
                     x304,  x305,  x306,  x307,  x308,  x309,  x310,  x311,  x312,  x313,  x314,
                     x315,  x316,  x317,  x318,  x319,  x320,  x321,  x322,  x323,  x324,  x1093,
                     x1094, x1095, x1096, x1097, x1098, x1099, x1100, x1101, x1102, x1103, x1104,
                     x1105, x1106, x1107, x1108, x1109, x1110, x1111, x1112, x1113, x1114, x1115,
                     x1116, x1117, x1118, x1119, x1120, x1121, x1122, x1123, x1124}},
      Val64Array5Array{
          Val64Array{x325,  x326,  x327,  x328,  x329,  x330,  x331,  x332,  x333,  x334,  x335,
                     x336,  x337,  x338,  x339,  x340,  x341,  x342,  x343,  x344,  x345,  x346,
                     x347,  x348,  x349,  x350,  x351,  x352,  x353,  x354,  x355,  x356,  x1125,
                     x1126, x1127, x1128, x1129, x1130, x1131, x1132, x1133, x1134, x1135, x1136,
                     x1137, x1138, x1139, x1140, x1141, x1142, x1143, x1144, x1145, x1146, x1147,
                     x1148, x1149, x1150, x1151, x1152, x1153, x1154, x1155, x1156},
          Val64Array{x357,  x358,  x359,  x360,  x361,  x362,  x363,  x364,  x365,  x366,  x367,
                     x368,  x369,  x370,  x371,  x372,  x373,  x374,  x375,  x376,  x377,  x378,
                     x379,  x380,  x381,  x382,  x383,  x384,  x385,  x386,  x387,  x388,  x1157,
                     x1158, x1159, x1160, x1161, x1162, x1163, x1164, x1165, x1166, x1167, x1168,
                     x1169, x1170, x1171, x1172, x1173, x1174, x1175, x1176, x1177, x1178, x1179,
                     x1180, x1181, x1182, x1183, x1184, x1185, x1186, x1187, x1188},
          Val64Array{x389,  x390,  x391,  x392,  x393,  x394,  x395,  x396,  x397,  x398,  x399,
                     x400,  x401,  x402,  x403,  x404,  x405,  x406,  x407,  x408,  x409,  x410,
                     x411,  x412,  x413,  x414,  x415,  x416,  x417,  x418,  x419,  x420,  x1189,
                     x1190, x1191, x1192, x1193, x1194, x1195, x1196, x1197, x1198, x1199, x1200,
                     x1201, x1202, x1203, x1204, x1205, x1206, x1207, x1208, x1209, x1210, x1211,
                     x1212, x1213, x1214, x1215, x1216, x1217, x1218, x1219, x1220},
          Val64Array{x421,  x422,  x423,  x424,  x425,  x426,  x427,  x428,  x429,  x430,  x431,
                     x432,  x433,  x434,  x435,  x436,  x437,  x438,  x439,  x440,  x441,  x442,
                     x443,  x444,  x445,  x446,  x447,  x448,  x449,  x450,  x451,  x452,  x1221,
                     x1222, x1223, x1224, x1225, x1226, x1227, x1228, x1229, x1230, x1231, x1232,
                     x1233, x1234, x1235, x1236, x1237, x1238, x1239, x1240, x1241, x1242, x1243,
                     x1244, x1245, x1246, x1247, x1248, x1249, x1250, x1251, x1252},
          Val64Array{x453,  x454,  x455,  x456,  x457,  x458,  x459,  x460,  x461,  x462,  x463,
                     x464,  x465,  x466,  x467,  x468,  x469,  x470,  x471,  x472,  x473,  x474,
                     x475,  x476,  x477,  x478,  x479,  x480,  x481,  x482,  x483,  x484,  x1253,
                     x1254, x1255, x1256, x1257, x1258, x1259, x1260, x1261, x1262, x1263, x1264,
                     x1265, x1266, x1267, x1268, x1269, x1270, x1271, x1272, x1273, x1274, x1275,
                     x1276, x1277, x1278, x1279, x1280, x1281, x1282, x1283, x1284}},
      Val64Array5Array{
          Val64Array{x485,  x486,  x487,  x488,  x489,  x490,  x491,  x492,  x493,  x494,  x495,
                     x496,  x497,  x498,  x499,  x500,  x501,  x502,  x503,  x504,  x505,  x506,
                     x507,  x508,  x509,  x510,  x511,  x512,  x513,  x514,  x515,  x516,  x1285,
                     x1286, x1287, x1288, x1289, x1290, x1291, x1292, x1293, x1294, x1295, x1296,
                     x1297, x1298, x1299, x1300, x1301, x1302, x1303, x1304, x1305, x1306, x1307,
                     x1308, x1309, x1310, x1311, x1312, x1313, x1314, x1315, x1316},
          Val64Array{x517,  x518,  x519,  x520,  x521,  x522,  x523,  x524,  x525,  x526,  x527,
                     x528,  x529,  x530,  x531,  x532,  x533,  x534,  x535,  x536,  x537,  x538,
                     x539,  x540,  x541,  x542,  x543,  x544,  x545,  x546,  x547,  x548,  x1317,
                     x1318, x1319, x1320, x1321, x1322, x1323, x1324, x1325, x1326, x1327, x1328,
                     x1329, x1330, x1331, x1332, x1333, x1334, x1335, x1336, x1337, x1338, x1339,
                     x1340, x1341, x1342, x1343, x1344, x1345, x1346, x1347, x1348},
          Val64Array{x549,  x550,  x551,  x552,  x553,  x554,  x555,  x556,  x557,  x558,  x559,
                     x560,  x561,  x562,  x563,  x564,  x565,  x566,  x567,  x568,  x569,  x570,
                     x571,  x572,  x573,  x574,  x575,  x576,  x577,  x578,  x579,  x580,  x1349,
                     x1350, x1351, x1352, x1353, x1354, x1355, x1356, x1357, x1358, x1359, x1360,
                     x1361, x1362, x1363, x1364, x1365, x1366, x1367, x1368, x1369, x1370, x1371,
                     x1372, x1373, x1374, x1375, x1376, x1377, x1378, x1379, x1380},
          Val64Array{x581,  x582,  x583,  x584,  x585,  x586,  x587,  x588,  x589,  x590,  x591,
                     x592,  x593,  x594,  x595,  x596,  x597,  x598,  x599,  x600,  x601,  x602,
                     x603,  x604,  x605,  x606,  x607,  x608,  x609,  x610,  x611,  x612,  x1381,
                     x1382, x1383, x1384, x1385, x1386, x1387, x1388, x1389, x1390, x1391, x1392,
                     x1393, x1394, x1395, x1396, x1397, x1398, x1399, x1400, x1401, x1402, x1403,
                     x1404, x1405, x1406, x1407, x1408, x1409, x1410, x1411, x1412},
          Val64Array{x613,  x614,  x615,  x616,  x617,  x618,  x619,  x620,  x621,  x622,  x623,
                     x624,  x625,  x626,  x627,  x628,  x629,  x630,  x631,  x632,  x633,  x634,
                     x635,  x636,  x637,  x638,  x639,  x640,  x641,  x642,  x643,  x644,  x1413,
                     x1414, x1415, x1416, x1417, x1418, x1419, x1420, x1421, x1422, x1423, x1424,
                     x1425, x1426, x1427, x1428, x1429, x1430, x1431, x1432, x1433, x1434, x1435,
                     x1436, x1437, x1438, x1439, x1440, x1441, x1442, x1443, x1444}},
      Val64Array5Array{
          Val64Array{x645,  x646,  x647,  x648,  x649,  x650,  x651,  x652,  x653,  x654,  x655,
                     x656,  x657,  x658,  x659,  x660,  x661,  x662,  x663,  x664,  x665,  x666,
                     x667,  x668,  x669,  x670,  x671,  x672,  x673,  x674,  x675,  x676,  x1445,
                     x1446, x1447, x1448, x1449, x1450, x1451, x1452, x1453, x1454, x1455, x1456,
                     x1457, x1458, x1459, x1460, x1461, x1462, x1463, x1464, x1465, x1466, x1467,
                     x1468, x1469, x1470, x1471, x1472, x1473, x1474, x1475, x1476},
          Val64Array{x677,  x678,  x679,  x680,  x681,  x682,  x683,  x684,  x685,  x686,  x687,
                     x688,  x689,  x690,  x691,  x692,  x693,  x694,  x695,  x696,  x697,  x698,
                     x699,  x700,  x701,  x702,  x703,  x704,  x705,  x706,  x707,  x708,  x1477,
                     x1478, x1479, x1480, x1481, x1482, x1483, x1484, x1485, x1486, x1487, x1488,
                     x1489, x1490, x1491, x1492, x1493, x1494, x1495, x1496, x1497, x1498, x1499,
                     x1500, x1501, x1502, x1503, x1504, x1505, x1506, x1507, x1508},
          Val64Array{x709,  x710,  x711,  x712,  x713,  x714,  x715,  x716,  x717,  x718,  x719,
                     x720,  x721,  x722,  x723,  x724,  x725,  x726,  x727,  x728,  x729,  x730,
                     x731,  x732,  x733,  x734,  x735,  x736,  x737,  x738,  x739,  x740,  x1509,
                     x1510, x1511, x1512, x1513, x1514, x1515, x1516, x1517, x1518, x1519, x1520,
                     x1521, x1522, x1523, x1524, x1525, x1526, x1527, x1528, x1529, x1530, x1531,
                     x1532, x1533, x1534, x1535, x1536, x1537, x1538, x1539, x1540},
          Val64Array{x741,  x742,  x743,  x744,  x745,  x746,  x747,  x748,  x749,  x750,  x751,
                     x752,  x753,  x754,  x755,  x756,  x757,  x758,  x759,  x760,  x761,  x762,
                     x763,  x764,  x765,  x766,  x767,  x768,  x769,  x770,  x771,  x772,  x1541,
                     x1542, x1543, x1544, x1545, x1546, x1547, x1548, x1549, x1550, x1551, x1552,
                     x1553, x1554, x1555, x1556, x1557, x1558, x1559, x1560, x1561, x1562, x1563,
                     x1564, x1565, x1566, x1567, x1568, x1569, x1570, x1571, x1572},
          Val64Array{x773,  x774,  x775,  x776,  x777,  x778,  x779,  x780,  x781,  x782,  x783,
                     x784,  x785,  x786,  x787,  x788,  x789,  x790,  x791,  x792,  x793,  x794,
                     x795,  x796,  x797,  x798,  x799,  x800,  x801,  x802,  x803,  x804,  x1573,
                     x1574, x1575, x1576, x1577, x1578, x1579, x1580, x1581, x1582, x1583, x1584,
                     x1585, x1586, x1587, x1588, x1589, x1590, x1591, x1592, x1593, x1594, x1595,
                     x1596, x1597, x1598, x1599, x1600, x1601, x1602, x1603, x1604}}};
  // KeccakRound0(zirgen/circuit/keccak2/top.zir:97)
  ThetaP1Struct x1606 = exec_ThetaP1(ctx, x1605, LAYOUT_LOOKUP(layout2, b));
  ThetaP1_Super_SuperStruct5Array x1607 = x1606._super;
  ThetaP1_Super_Super_SuperStruct64Array x1608 = x1607[0]._super;
  Val x1609 = x1608[0]._super._super;
  Val x1610 = x1608[1]._super._super;
  Val x1611 = x1608[2]._super._super;
  Val x1612 = x1608[3]._super._super;
  Val x1613 = x1608[4]._super._super;
  Val x1614 = x1608[5]._super._super;
  Val x1615 = x1608[6]._super._super;
  Val x1616 = x1608[7]._super._super;
  Val x1617 = x1608[8]._super._super;
  Val x1618 = x1608[9]._super._super;
  Val x1619 = x1608[10]._super._super;
  Val x1620 = x1608[11]._super._super;
  Val x1621 = x1608[12]._super._super;
  Val x1622 = x1608[13]._super._super;
  Val x1623 = x1608[14]._super._super;
  Val x1624 = x1608[15]._super._super;
  Val x1625 = x1608[16]._super._super;
  Val x1626 = x1608[17]._super._super;
  Val x1627 = x1608[18]._super._super;
  Val x1628 = x1608[19]._super._super;
  Val x1629 = x1608[20]._super._super;
  Val x1630 = x1608[21]._super._super;
  Val x1631 = x1608[22]._super._super;
  Val x1632 = x1608[23]._super._super;
  Val x1633 = x1608[24]._super._super;
  Val x1634 = x1608[25]._super._super;
  Val x1635 = x1608[26]._super._super;
  Val x1636 = x1608[27]._super._super;
  Val x1637 = x1608[28]._super._super;
  Val x1638 = x1608[29]._super._super;
  Val x1639 = x1608[30]._super._super;
  Val x1640 = x1608[31]._super._super;
  Val x1641 = x1608[32]._super._super;
  Val x1642 = x1608[33]._super._super;
  Val x1643 = x1608[34]._super._super;
  Val x1644 = x1608[35]._super._super;
  Val x1645 = x1608[36]._super._super;
  Val x1646 = x1608[37]._super._super;
  Val x1647 = x1608[38]._super._super;
  Val x1648 = x1608[39]._super._super;
  Val x1649 = x1608[40]._super._super;
  Val x1650 = x1608[41]._super._super;
  Val x1651 = x1608[42]._super._super;
  Val x1652 = x1608[43]._super._super;
  Val x1653 = x1608[44]._super._super;
  Val x1654 = x1608[45]._super._super;
  Val x1655 = x1608[46]._super._super;
  Val x1656 = x1608[47]._super._super;
  Val x1657 = x1608[48]._super._super;
  Val x1658 = x1608[49]._super._super;
  Val x1659 = x1608[50]._super._super;
  Val x1660 = x1608[51]._super._super;
  Val x1661 = x1608[52]._super._super;
  Val x1662 = x1608[53]._super._super;
  Val x1663 = x1608[54]._super._super;
  Val x1664 = x1608[55]._super._super;
  Val x1665 = x1608[56]._super._super;
  Val x1666 = x1608[57]._super._super;
  Val x1667 = x1608[58]._super._super;
  Val x1668 = x1608[59]._super._super;
  Val x1669 = x1608[60]._super._super;
  Val x1670 = x1608[61]._super._super;
  Val x1671 = x1608[62]._super._super;
  Val x1672 = x1608[63]._super._super;
  ThetaP1_Super_Super_SuperStruct64Array x1673 = x1607[1]._super;
  Val x1674 = x1673[0]._super._super;
  Val x1675 = x1673[1]._super._super;
  Val x1676 = x1673[2]._super._super;
  Val x1677 = x1673[3]._super._super;
  Val x1678 = x1673[4]._super._super;
  Val x1679 = x1673[5]._super._super;
  Val x1680 = x1673[6]._super._super;
  Val x1681 = x1673[7]._super._super;
  Val x1682 = x1673[8]._super._super;
  Val x1683 = x1673[9]._super._super;
  Val x1684 = x1673[10]._super._super;
  Val x1685 = x1673[11]._super._super;
  Val x1686 = x1673[12]._super._super;
  Val x1687 = x1673[13]._super._super;
  Val x1688 = x1673[14]._super._super;
  Val x1689 = x1673[15]._super._super;
  Val x1690 = x1673[16]._super._super;
  Val x1691 = x1673[17]._super._super;
  Val x1692 = x1673[18]._super._super;
  Val x1693 = x1673[19]._super._super;
  Val x1694 = x1673[20]._super._super;
  Val x1695 = x1673[21]._super._super;
  Val x1696 = x1673[22]._super._super;
  Val x1697 = x1673[23]._super._super;
  Val x1698 = x1673[24]._super._super;
  Val x1699 = x1673[25]._super._super;
  Val x1700 = x1673[26]._super._super;
  Val x1701 = x1673[27]._super._super;
  Val x1702 = x1673[28]._super._super;
  Val x1703 = x1673[29]._super._super;
  Val x1704 = x1673[30]._super._super;
  Val x1705 = x1673[31]._super._super;
  Val x1706 = x1673[32]._super._super;
  Val x1707 = x1673[33]._super._super;
  Val x1708 = x1673[34]._super._super;
  Val x1709 = x1673[35]._super._super;
  Val x1710 = x1673[36]._super._super;
  Val x1711 = x1673[37]._super._super;
  Val x1712 = x1673[38]._super._super;
  Val x1713 = x1673[39]._super._super;
  Val x1714 = x1673[40]._super._super;
  Val x1715 = x1673[41]._super._super;
  Val x1716 = x1673[42]._super._super;
  Val x1717 = x1673[43]._super._super;
  Val x1718 = x1673[44]._super._super;
  Val x1719 = x1673[45]._super._super;
  Val x1720 = x1673[46]._super._super;
  Val x1721 = x1673[47]._super._super;
  Val x1722 = x1673[48]._super._super;
  Val x1723 = x1673[49]._super._super;
  Val x1724 = x1673[50]._super._super;
  Val x1725 = x1673[51]._super._super;
  Val x1726 = x1673[52]._super._super;
  Val x1727 = x1673[53]._super._super;
  Val x1728 = x1673[54]._super._super;
  Val x1729 = x1673[55]._super._super;
  Val x1730 = x1673[56]._super._super;
  Val x1731 = x1673[57]._super._super;
  Val x1732 = x1673[58]._super._super;
  Val x1733 = x1673[59]._super._super;
  Val x1734 = x1673[60]._super._super;
  Val x1735 = x1673[61]._super._super;
  Val x1736 = x1673[62]._super._super;
  Val x1737 = x1673[63]._super._super;
  ThetaP1_Super_Super_SuperStruct64Array x1738 = x1607[2]._super;
  Val x1739 = x1738[0]._super._super;
  Val x1740 = x1738[1]._super._super;
  Val x1741 = x1738[2]._super._super;
  Val x1742 = x1738[3]._super._super;
  Val x1743 = x1738[4]._super._super;
  Val x1744 = x1738[5]._super._super;
  Val x1745 = x1738[6]._super._super;
  Val x1746 = x1738[7]._super._super;
  Val x1747 = x1738[8]._super._super;
  Val x1748 = x1738[9]._super._super;
  Val x1749 = x1738[10]._super._super;
  Val x1750 = x1738[11]._super._super;
  Val x1751 = x1738[12]._super._super;
  Val x1752 = x1738[13]._super._super;
  Val x1753 = x1738[14]._super._super;
  Val x1754 = x1738[15]._super._super;
  Val x1755 = x1738[16]._super._super;
  Val x1756 = x1738[17]._super._super;
  Val x1757 = x1738[18]._super._super;
  Val x1758 = x1738[19]._super._super;
  Val x1759 = x1738[20]._super._super;
  Val x1760 = x1738[21]._super._super;
  Val x1761 = x1738[22]._super._super;
  Val x1762 = x1738[23]._super._super;
  Val x1763 = x1738[24]._super._super;
  Val x1764 = x1738[25]._super._super;
  Val x1765 = x1738[26]._super._super;
  Val x1766 = x1738[27]._super._super;
  Val x1767 = x1738[28]._super._super;
  Val x1768 = x1738[29]._super._super;
  Val x1769 = x1738[30]._super._super;
  Val x1770 = x1738[31]._super._super;
  Val x1771 = x1738[32]._super._super;
  Val x1772 = x1738[33]._super._super;
  Val x1773 = x1738[34]._super._super;
  Val x1774 = x1738[35]._super._super;
  Val x1775 = x1738[36]._super._super;
  Val x1776 = x1738[37]._super._super;
  Val x1777 = x1738[38]._super._super;
  Val x1778 = x1738[39]._super._super;
  Val x1779 = x1738[40]._super._super;
  Val x1780 = x1738[41]._super._super;
  Val x1781 = x1738[42]._super._super;
  Val x1782 = x1738[43]._super._super;
  Val x1783 = x1738[44]._super._super;
  Val x1784 = x1738[45]._super._super;
  Val x1785 = x1738[46]._super._super;
  Val x1786 = x1738[47]._super._super;
  Val x1787 = x1738[48]._super._super;
  Val x1788 = x1738[49]._super._super;
  Val x1789 = x1738[50]._super._super;
  Val x1790 = x1738[51]._super._super;
  Val x1791 = x1738[52]._super._super;
  Val x1792 = x1738[53]._super._super;
  Val x1793 = x1738[54]._super._super;
  Val x1794 = x1738[55]._super._super;
  Val x1795 = x1738[56]._super._super;
  Val x1796 = x1738[57]._super._super;
  Val x1797 = x1738[58]._super._super;
  Val x1798 = x1738[59]._super._super;
  Val x1799 = x1738[60]._super._super;
  Val x1800 = x1738[61]._super._super;
  Val x1801 = x1738[62]._super._super;
  Val x1802 = x1738[63]._super._super;
  ThetaP1_Super_Super_SuperStruct64Array x1803 = x1607[3]._super;
  Val x1804 = x1803[0]._super._super;
  Val x1805 = x1803[1]._super._super;
  Val x1806 = x1803[2]._super._super;
  Val x1807 = x1803[3]._super._super;
  Val x1808 = x1803[4]._super._super;
  Val x1809 = x1803[5]._super._super;
  Val x1810 = x1803[6]._super._super;
  Val x1811 = x1803[7]._super._super;
  Val x1812 = x1803[8]._super._super;
  Val x1813 = x1803[9]._super._super;
  Val x1814 = x1803[10]._super._super;
  Val x1815 = x1803[11]._super._super;
  Val x1816 = x1803[12]._super._super;
  Val x1817 = x1803[13]._super._super;
  Val x1818 = x1803[14]._super._super;
  Val x1819 = x1803[15]._super._super;
  Val x1820 = x1803[16]._super._super;
  Val x1821 = x1803[17]._super._super;
  Val x1822 = x1803[18]._super._super;
  Val x1823 = x1803[19]._super._super;
  Val x1824 = x1803[20]._super._super;
  Val x1825 = x1803[21]._super._super;
  Val x1826 = x1803[22]._super._super;
  Val x1827 = x1803[23]._super._super;
  Val x1828 = x1803[24]._super._super;
  Val x1829 = x1803[25]._super._super;
  Val x1830 = x1803[26]._super._super;
  Val x1831 = x1803[27]._super._super;
  Val x1832 = x1803[28]._super._super;
  Val x1833 = x1803[29]._super._super;
  Val x1834 = x1803[30]._super._super;
  Val x1835 = x1803[31]._super._super;
  Val x1836 = x1803[32]._super._super;
  Val x1837 = x1803[33]._super._super;
  Val x1838 = x1803[34]._super._super;
  Val x1839 = x1803[35]._super._super;
  Val x1840 = x1803[36]._super._super;
  Val x1841 = x1803[37]._super._super;
  Val x1842 = x1803[38]._super._super;
  Val x1843 = x1803[39]._super._super;
  Val x1844 = x1803[40]._super._super;
  Val x1845 = x1803[41]._super._super;
  Val x1846 = x1803[42]._super._super;
  Val x1847 = x1803[43]._super._super;
  Val x1848 = x1803[44]._super._super;
  Val x1849 = x1803[45]._super._super;
  Val x1850 = x1803[46]._super._super;
  Val x1851 = x1803[47]._super._super;
  Val x1852 = x1803[48]._super._super;
  Val x1853 = x1803[49]._super._super;
  Val x1854 = x1803[50]._super._super;
  Val x1855 = x1803[51]._super._super;
  Val x1856 = x1803[52]._super._super;
  Val x1857 = x1803[53]._super._super;
  Val x1858 = x1803[54]._super._super;
  Val x1859 = x1803[55]._super._super;
  Val x1860 = x1803[56]._super._super;
  Val x1861 = x1803[57]._super._super;
  Val x1862 = x1803[58]._super._super;
  Val x1863 = x1803[59]._super._super;
  Val x1864 = x1803[60]._super._super;
  Val x1865 = x1803[61]._super._super;
  Val x1866 = x1803[62]._super._super;
  Val x1867 = x1803[63]._super._super;
  ThetaP1_Super_Super_SuperStruct64Array x1868 = x1607[4]._super;
  Val x1869 = x1868[0]._super._super;
  Val x1870 = x1868[1]._super._super;
  Val x1871 = x1868[2]._super._super;
  Val x1872 = x1868[3]._super._super;
  Val x1873 = x1868[4]._super._super;
  Val x1874 = x1868[5]._super._super;
  Val x1875 = x1868[6]._super._super;
  Val x1876 = x1868[7]._super._super;
  Val x1877 = x1868[8]._super._super;
  Val x1878 = x1868[9]._super._super;
  Val x1879 = x1868[10]._super._super;
  Val x1880 = x1868[11]._super._super;
  Val x1881 = x1868[12]._super._super;
  Val x1882 = x1868[13]._super._super;
  Val x1883 = x1868[14]._super._super;
  Val x1884 = x1868[15]._super._super;
  Val x1885 = x1868[16]._super._super;
  Val x1886 = x1868[17]._super._super;
  Val x1887 = x1868[18]._super._super;
  Val x1888 = x1868[19]._super._super;
  Val x1889 = x1868[20]._super._super;
  Val x1890 = x1868[21]._super._super;
  Val x1891 = x1868[22]._super._super;
  Val x1892 = x1868[23]._super._super;
  Val x1893 = x1868[24]._super._super;
  Val x1894 = x1868[25]._super._super;
  Val x1895 = x1868[26]._super._super;
  Val x1896 = x1868[27]._super._super;
  Val x1897 = x1868[28]._super._super;
  Val x1898 = x1868[29]._super._super;
  Val x1899 = x1868[30]._super._super;
  Val x1900 = x1868[31]._super._super;
  Val x1901 = x1868[32]._super._super;
  Val x1902 = x1868[33]._super._super;
  Val x1903 = x1868[34]._super._super;
  Val x1904 = x1868[35]._super._super;
  Val x1905 = x1868[36]._super._super;
  Val x1906 = x1868[37]._super._super;
  Val x1907 = x1868[38]._super._super;
  Val x1908 = x1868[39]._super._super;
  Val x1909 = x1868[40]._super._super;
  Val x1910 = x1868[41]._super._super;
  Val x1911 = x1868[42]._super._super;
  Val x1912 = x1868[43]._super._super;
  Val x1913 = x1868[44]._super._super;
  Val x1914 = x1868[45]._super._super;
  Val x1915 = x1868[46]._super._super;
  Val x1916 = x1868[47]._super._super;
  Val x1917 = x1868[48]._super._super;
  Val x1918 = x1868[49]._super._super;
  Val x1919 = x1868[50]._super._super;
  Val x1920 = x1868[51]._super._super;
  Val x1921 = x1868[52]._super._super;
  Val x1922 = x1868[53]._super._super;
  Val x1923 = x1868[54]._super._super;
  Val x1924 = x1868[55]._super._super;
  Val x1925 = x1868[56]._super._super;
  Val x1926 = x1868[57]._super._super;
  Val x1927 = x1868[58]._super._super;
  Val x1928 = x1868[59]._super._super;
  Val x1929 = x1868[60]._super._super;
  Val x1930 = x1868[61]._super._super;
  Val x1931 = x1868[62]._super._super;
  Val x1932 = x1868[63]._super._super;
  // KeccakRound0(zirgen/circuit/keccak2/top.zir:94)
  TopStateKflatStruct100Array x1933 = arg1.kflat;
  NondetRegStruct16Array x1934 = arg1.sflat;
  Val x1935 = x1933[0]._super._super;
  Val x1936 = x1933[1]._super._super;
  Val x1937 = x1933[2]._super._super;
  Val x1938 = x1933[3]._super._super;
  Val x1939 = x1933[4]._super._super;
  Val x1940 = x1933[5]._super._super;
  Val x1941 = x1933[6]._super._super;
  Val x1942 = x1933[7]._super._super;
  Val x1943 = x1933[8]._super._super;
  Val x1944 = x1933[9]._super._super;
  Val x1945 = x1933[10]._super._super;
  Val x1946 = x1933[11]._super._super;
  Val x1947 = x1933[12]._super._super;
  Val x1948 = x1933[13]._super._super;
  Val x1949 = x1933[14]._super._super;
  Val x1950 = x1933[15]._super._super;
  Val x1951 = x1933[16]._super._super;
  Val x1952 = x1933[17]._super._super;
  Val x1953 = x1933[18]._super._super;
  Val x1954 = x1933[19]._super._super;
  Val x1955 = x1933[20]._super._super;
  Val x1956 = x1933[21]._super._super;
  Val x1957 = x1933[22]._super._super;
  Val x1958 = x1933[23]._super._super;
  Val x1959 = x1933[24]._super._super;
  Val x1960 = x1933[25]._super._super;
  Val x1961 = x1933[26]._super._super;
  Val x1962 = x1933[27]._super._super;
  Val x1963 = x1933[28]._super._super;
  Val x1964 = x1933[29]._super._super;
  Val x1965 = x1933[30]._super._super;
  Val x1966 = x1933[31]._super._super;
  Val x1967 = x1933[32]._super._super;
  Val x1968 = x1933[33]._super._super;
  Val x1969 = x1933[34]._super._super;
  Val x1970 = x1933[35]._super._super;
  Val x1971 = x1933[36]._super._super;
  Val x1972 = x1933[37]._super._super;
  Val x1973 = x1933[38]._super._super;
  Val x1974 = x1933[39]._super._super;
  Val x1975 = x1933[40]._super._super;
  Val x1976 = x1933[41]._super._super;
  Val x1977 = x1933[42]._super._super;
  Val x1978 = x1933[43]._super._super;
  Val x1979 = x1933[44]._super._super;
  Val x1980 = x1933[45]._super._super;
  Val x1981 = x1933[46]._super._super;
  Val x1982 = x1933[47]._super._super;
  Val x1983 = x1933[48]._super._super;
  Val x1984 = x1933[49]._super._super;
  Val x1985 = x1933[50]._super._super;
  Val x1986 = x1933[51]._super._super;
  Val x1987 = x1933[52]._super._super;
  Val x1988 = x1933[53]._super._super;
  Val x1989 = x1933[54]._super._super;
  Val x1990 = x1933[55]._super._super;
  Val x1991 = x1933[56]._super._super;
  Val x1992 = x1933[57]._super._super;
  Val x1993 = x1933[58]._super._super;
  Val x1994 = x1933[59]._super._super;
  Val x1995 = x1933[60]._super._super;
  Val x1996 = x1933[61]._super._super;
  Val x1997 = x1933[62]._super._super;
  Val x1998 = x1933[63]._super._super;
  Val x1999 = x1933[64]._super._super;
  Val x2000 = x1933[65]._super._super;
  Val x2001 = x1933[66]._super._super;
  Val x2002 = x1933[67]._super._super;
  Val x2003 = x1933[68]._super._super;
  Val x2004 = x1933[69]._super._super;
  Val x2005 = x1933[70]._super._super;
  Val x2006 = x1933[71]._super._super;
  Val x2007 = x1933[72]._super._super;
  Val x2008 = x1933[73]._super._super;
  Val x2009 = x1933[74]._super._super;
  Val x2010 = x1933[75]._super._super;
  Val x2011 = x1933[76]._super._super;
  Val x2012 = x1933[77]._super._super;
  Val x2013 = x1933[78]._super._super;
  Val x2014 = x1933[79]._super._super;
  Val x2015 = x1933[80]._super._super;
  Val x2016 = x1933[81]._super._super;
  Val x2017 = x1933[82]._super._super;
  Val x2018 = x1933[83]._super._super;
  Val x2019 = x1933[84]._super._super;
  Val x2020 = x1933[85]._super._super;
  Val x2021 = x1933[86]._super._super;
  Val x2022 = x1933[87]._super._super;
  Val x2023 = x1933[88]._super._super;
  Val x2024 = x1933[89]._super._super;
  Val x2025 = x1933[90]._super._super;
  Val x2026 = x1933[91]._super._super;
  Val x2027 = x1933[92]._super._super;
  Val x2028 = x1933[93]._super._super;
  Val x2029 = x1933[94]._super._super;
  Val x2030 = x1933[95]._super._super;
  Val x2031 = x1933[96]._super._super;
  Val x2032 = x1933[97]._super._super;
  Val x2033 = x1933[98]._super._super;
  Val x2034 = x1933[99]._super._super;
  Val16Array x2035 = Val16Array{x1934[0]._super,
                                x1934[1]._super,
                                x1934[2]._super,
                                x1934[3]._super,
                                x1934[4]._super,
                                x1934[5]._super,
                                x1934[6]._super,
                                x1934[7]._super,
                                x1934[8]._super,
                                x1934[9]._super,
                                x1934[10]._super,
                                x1934[11]._super,
                                x1934[12]._super,
                                x1934[13]._super,
                                x1934[14]._super,
                                x1934[15]._super};
  // KeccakRound0(zirgen/circuit/keccak2/top.zir:98)
  TopStateStruct x2036 = exec_TopState(
      ctx,
      Val800Array{
          x1609,  x1610,  x1611,  x1612,  x1613,  x1614,  x1615,  x1616,  x1617,  x1618,  x1619,
          x1620,  x1621,  x1622,  x1623,  x1624,  x1625,  x1626,  x1627,  x1628,  x1629,  x1630,
          x1631,  x1632,  x1633,  x1634,  x1635,  x1636,  x1637,  x1638,  x1639,  x1640,  x1641,
          x1642,  x1643,  x1644,  x1645,  x1646,  x1647,  x1648,  x1649,  x1650,  x1651,  x1652,
          x1653,  x1654,  x1655,  x1656,  x1657,  x1658,  x1659,  x1660,  x1661,  x1662,  x1663,
          x1664,  x1665,  x1666,  x1667,  x1668,  x1669,  x1670,  x1671,  x1672,  x1674,  x1675,
          x1676,  x1677,  x1678,  x1679,  x1680,  x1681,  x1682,  x1683,  x1684,  x1685,  x1686,
          x1687,  x1688,  x1689,  x1690,  x1691,  x1692,  x1693,  x1694,  x1695,  x1696,  x1697,
          x1698,  x1699,  x1700,  x1701,  x1702,  x1703,  x1704,  x1705,  x1706,  x1707,  x1708,
          x1709,  x1710,  x1711,  x1712,  x1713,  x1714,  x1715,  x1716,  x1717,  x1718,  x1719,
          x1720,  x1721,  x1722,  x1723,  x1724,  x1725,  x1726,  x1727,  x1728,  x1729,  x1730,
          x1731,  x1732,  x1733,  x1734,  x1735,  x1736,  x1737,  x1739,  x1740,  x1741,  x1742,
          x1743,  x1744,  x1745,  x1746,  x1747,  x1748,  x1749,  x1750,  x1751,  x1752,  x1753,
          x1754,  x1755,  x1756,  x1757,  x1758,  x1759,  x1760,  x1761,  x1762,  x1763,  x1764,
          x1765,  x1766,  x1767,  x1768,  x1769,  x1770,  x1771,  x1772,  x1773,  x1774,  x1775,
          x1776,  x1777,  x1778,  x1779,  x1780,  x1781,  x1782,  x1783,  x1784,  x1785,  x1786,
          x1787,  x1788,  x1789,  x1790,  x1791,  x1792,  x1793,  x1794,  x1795,  x1796,  x1797,
          x1798,  x1799,  x1800,  x1801,  x1802,  x1804,  x1805,  x1806,  x1807,  x1808,  x1809,
          x1810,  x1811,  x1812,  x1813,  x1814,  x1815,  x1816,  x1817,  x1818,  x1819,  x1820,
          x1821,  x1822,  x1823,  x1824,  x1825,  x1826,  x1827,  x1828,  x1829,  x1830,  x1831,
          x1832,  x1833,  x1834,  x1835,  x1836,  x1837,  x1838,  x1839,  x1840,  x1841,  x1842,
          x1843,  x1844,  x1845,  x1846,  x1847,  x1848,  x1849,  x1850,  x1851,  x1852,  x1853,
          x1854,  x1855,  x1856,  x1857,  x1858,  x1859,  x1860,  x1861,  x1862,  x1863,  x1864,
          x1865,  x1866,  x1867,  x1869,  x1870,  x1871,  x1872,  x1873,  x1874,  x1875,  x1876,
          x1877,  x1878,  x1879,  x1880,  x1881,  x1882,  x1883,  x1884,  x1885,  x1886,  x1887,
          x1888,  x1889,  x1890,  x1891,  x1892,  x1893,  x1894,  x1895,  x1896,  x1897,  x1898,
          x1899,  x1900,  x1901,  x1902,  x1903,  x1904,  x1905,  x1906,  x1907,  x1908,  x1909,
          x1910,  x1911,  x1912,  x1913,  x1914,  x1915,  x1916,  x1917,  x1918,  x1919,  x1920,
          x1921,  x1922,  x1923,  x1924,  x1925,  x1926,  x1927,  x1928,  x1929,  x1930,  x1931,
          x1932,  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val100Array{
          x1935, x1936, x1937, x1938, x1939, x1940, x1941, x1942, x1943, x1944, x1945, x1946, x1947,
          x1948, x1949, x1950, x1951, x1952, x1953, x1954, x1955, x1956, x1957, x1958, x1959, x1960,
          x1961, x1962, x1963, x1964, x1965, x1966, x1967, x1968, x1969, x1970, x1971, x1972, x1973,
          x1974, x1975, x1976, x1977, x1978, x1979, x1980, x1981, x1982, x1983, x1984, x1985, x1986,
          x1987, x1988, x1989, x1990, x1991, x1992, x1993, x1994, x1995, x1996, x1997, x1998, x1999,
          x2000, x2001, x2002, x2003, x2004, x2005, x2006, x2007, x2008, x2009, x2010, x2011, x2012,
          x2013, x2014, x2015, x2016, x2017, x2018, x2019, x2020, x2021, x2022, x2023, x2024, x2025,
          x2026, x2027, x2028, x2029, x2030, x2031, x2032, x2033, x2034},
      x2035,
      LAYOUT_LOOKUP(layout2, _super));
  return x2036;
}

} // namespace risc0::impl
