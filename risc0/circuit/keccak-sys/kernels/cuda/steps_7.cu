#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ ThetaP1Struct exec_ThetaP1(ExecContext& ctx,
                                      Val64Array5Array5Array arg0,
                                      BoundLayout<ThetaP1Layout> layout1) {
  // ThetaP1(zirgen/circuit/keccak2/keccak.zir:11)
  ThetaP1_Super_SuperStruct5Array x2 =
      map(Val5Array{Val(0), Val(1), Val(2), Val(3), Val(4)},
          LAYOUT_LOOKUP(layout1, _super),
          ([&](Val5Array::value_type x3,
               BoundLayout<ThetaP1_Super_SuperLayout5LayoutArray::value_type> x4) {
            // ThetaP1(zirgen/circuit/keccak2/keccak.zir:12)
            ThetaP1_Super_Super_SuperStruct64Array x5 = map(
                Val64Array{Val(0),  Val(1),  Val(2),  Val(3),  Val(4),  Val(5),  Val(6),  Val(7),
                           Val(8),  Val(9),  Val(10), Val(11), Val(12), Val(13), Val(14), Val(15),
                           Val(16), Val(17), Val(18), Val(19), Val(20), Val(21), Val(22), Val(23),
                           Val(24), Val(25), Val(26), Val(27), Val(28), Val(29), Val(30), Val(31),
                           Val(32), Val(33), Val(34), Val(35), Val(36), Val(37), Val(38), Val(39),
                           Val(40), Val(41), Val(42), Val(43), Val(44), Val(45), Val(46), Val(47),
                           Val(48), Val(49), Val(50), Val(51), Val(52), Val(53), Val(54), Val(55),
                           Val(56), Val(57), Val(58), Val(59), Val(60), Val(61), Val(62), Val(63)},
                LAYOUT_LOOKUP(x4, _super),
                ([&](Val64Array::value_type x6,
                     BoundLayout<NondetRegLayout64LayoutArray::value_type> x7) {
                  // ThetaP1(zirgen/circuit/keccak2/keccak.zir:13)
                  Val5Array x8 = Val5Array{arg0[0][to_size_t(x3)][to_size_t(x6)],
                                           arg0[1][to_size_t(x3)][to_size_t(x6)],
                                           arg0[2][to_size_t(x3)][to_size_t(x6)],
                                           arg0[3][to_size_t(x3)][to_size_t(x6)],
                                           arg0[4][to_size_t(x3)][to_size_t(x6)]};
                  NondetRegStruct x9 = exec_Xor5(ctx, x8, x7);
                  return ThetaP1_Super_Super_SuperStruct{._super = x9};
                }));
            return ThetaP1_Super_SuperStruct{._super = x5};
          }));
  return ThetaP1Struct{._super = x2};
}
__device__ TopStateStruct exec_InitCycle(ExecContext& ctx, BoundLayout<TopStateLayout> layout0) {
  // Log(<preamble>:22)
  // InitCycle(zirgen/circuit/keccak2/top.zir:307)
  INVOKE_EXTERN(ctx, log, "InitCycle", std::initializer_list<Val>{});
  // InitCycle(zirgen/circuit/keccak2/top.zir:316)
  TopStateStruct x1 = exec_TopState(
      ctx,
      Val800Array{
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val100Array{Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val16Array{Val(58983),
                 Val(27145),
                 Val(44677),
                 Val(47975),
                 Val(62322),
                 Val(15470),
                 Val(62778),
                 Val(42319),
                 Val(21119),
                 Val(20750),
                 Val(26764),
                 Val(39685),
                 Val(55723),
                 Val(8067),
                 Val(52505),
                 Val(23520)},
      layout0);
  return x1;
}
__device__ WrapOneHotStruct exec_WrapOneHot(ExecContext& ctx,
                                            OneHot_12_Struct arg0,
                                            BoundLayout<WrapOneHotLayout> layout1) {
  // WrapOneHot(zirgen/circuit/keccak2/top.zir:470)
  WrapOneHot_SuperStruct12Array x2 = map(
      Val12Array{Val(0),
                 Val(1),
                 Val(2),
                 Val(3),
                 Val(4),
                 Val(5),
                 Val(6),
                 Val(7),
                 Val(8),
                 Val(9),
                 Val(10),
                 Val(11)},
      LAYOUT_LOOKUP(layout1, _super),
      ([&](Val12Array::value_type x3, BoundLayout<NondetRegLayout12LayoutArray::value_type> x4) {
        NondetRegStruct x5 = exec_Reg(ctx, arg0.bits[to_size_t(x3)]._super._super, x4);
        return WrapOneHot_SuperStruct{._super = x5};
      }));
  return WrapOneHotStruct{._super = x2};
}

} // namespace risc0::circuit::keccak::cuda
