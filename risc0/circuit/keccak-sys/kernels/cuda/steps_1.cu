#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ NondetRegStruct exec_NondetReg(ExecContext& ctx,
                                          Val arg0,
                                          BoundLayout<NondetRegLayout> layout1) {
  STORE(LAYOUT_LOOKUP(layout1, _super), arg0);
  NondetRegStruct x2 = NondetRegStruct{._super = LOAD(LAYOUT_LOOKUP(layout1, _super), 0)};
  return x2;
}
__device__ OneHot_8_Struct exec_OneHot_8_(ExecContext& ctx,
                                          Val arg0,
                                          BoundLayout<OneHot_8_Layout> layout1) {
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:7)
  NondetRegStruct8Array x2 =
      map(Val8Array{Val(0), Val(1), Val(2), Val(3), Val(4), Val(5), Val(6), Val(7)},
          LAYOUT_LOOKUP(layout1, _super),
          ([&](Val8Array::value_type x3, BoundLayout<NondetRegLayout8LayoutArray::value_type> x4) {
            NondetRegStruct x5 = exec_NondetBitReg(ctx, isz((x3 - arg0)), x4);
            return x5;
          }));
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:9)
  Val x6 = (x2[0]._super + x2[1]._super);
  Val x7 = ((x6 + x2[2]._super) + x2[3]._super);
  Val x8 = ((x7 + x2[4]._super) + x2[5]._super);
  Val x9 = ((x8 + x2[6]._super) + x2[7]._super);
  EQZ((x9 - Val(1)), "OneHot(zirgen/circuit/keccak2/one_hot.zir:9)");
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:11)
  Val x10 = (x2[2]._super * Val(2));
  Val x11 = (x2[3]._super * Val(3));
  Val x12 = (x2[4]._super * Val(4));
  Val x13 = (x2[5]._super * Val(5));
  Val x14 = (x2[6]._super * Val(6));
  Val x15 = (x2[7]._super * Val(7));
  Val x16 = (x2[1]._super + x10);
  Val x17 = (((x16 + x11) + x12) + x13);
  Val x18 = (((x17 + x14) + x15) - arg0);
  EQZ(x18, "OneHot(zirgen/circuit/keccak2/one_hot.zir:11)");
  return OneHot_8_Struct{._super = x2};
}
__device__ ControlStateStruct exec_KeccackNextRound(ExecContext& ctx,
                                                    ControlStateStruct arg0,
                                                    BoundLayout<KeccackNextRoundLayout> layout1) {
  // KeccackNextRound(zirgen/circuit/keccak2/top.zir:410)
  Val x2 = (arg0.round._super - Val(23));
  NondetRegStruct x3 = exec_IsZero(ctx, x2, LAYOUT_LOOKUP(layout1, isLast));
  // KeccackNextRound(zirgen/circuit/keccak2/top.zir:414)
  Val x4 = (arg0.round._super + Val(1));
  ControlStateStruct x5;
  if (to_size_t(x3._super)) {
    // KeccackNextRound(zirgen/circuit/keccak2/top.zir:412)
    ControlStateStruct x6 =
        exec_ControlState(ctx, Val(3), Val(0), Val(0), Val(0), LAYOUT_LOOKUP(layout1, _super.arm0));
    x5 = x6;
  } else if (to_size_t((Val(1) - x3._super))) {
    // KeccackNextRound(zirgen/circuit/keccak2/top.zir:414)
    ControlStateStruct x7 =
        exec_ControlState(ctx, Val(4), Val(0), Val(0), x4, LAYOUT_LOOKUP(layout1, _super.arm1));
    x5 = x7;
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  // KeccackNextRound(zirgen/circuit/keccak2/top.zir:411)
  ControlStateStruct x8 = back_ControlState(ctx, 0, LAYOUT_LOOKUP(layout1, _super._super));
  return x8;
}

} // namespace risc0::circuit::keccak::cuda
