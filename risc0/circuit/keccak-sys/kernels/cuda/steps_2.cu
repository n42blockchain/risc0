#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ NondetRegStruct back_Reg(ExecContext& ctx,
                                    Index distance0,
                                    BoundLayout<NondetRegLayout> layout1) {
  // Reg(<preamble>:4)
  NondetRegStruct x2 = back_NondetReg(ctx, distance0, layout1);
  return x2;
}
__device__ DoShaStepStruct exec_DoShaStep(ExecContext& ctx,
                                          ShaStateStruct arg0,
                                          Val2Array arg1,
                                          Val arg2,
                                          Val2Array arg3,
                                          BoundLayout<DoShaStepLayout> layout4) {
  // BitXor(zirgen/circuit/keccak2/bits.zir:32)
  // XorU32(zirgen/circuit/keccak2/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak2/sha2.zir:59)
  // DoShaStep(zirgen/circuit/keccak2/top.zir:137)
  Val x5 = (arg0.w[14][18] + arg0.w[14][3]);
  Val x6 = (arg0.w[14][18] * Val(2));
  Val x7 = (x5 - (x6 * arg0.w[14][3]));
  Val x8 = (arg0.w[14][19] + arg0.w[14][4]);
  Val x9 = (arg0.w[14][19] * Val(2));
  Val x10 = (x8 - (x9 * arg0.w[14][4]));
  Val x11 = (arg0.w[14][20] + arg0.w[14][5]);
  Val x12 = (arg0.w[14][20] * Val(2));
  Val x13 = (x11 - (x12 * arg0.w[14][5]));
  Val x14 = (arg0.w[14][21] + arg0.w[14][6]);
  Val x15 = (arg0.w[14][21] * Val(2));
  Val x16 = (x14 - (x15 * arg0.w[14][6]));
  Val x17 = (arg0.w[14][22] + arg0.w[14][7]);
  Val x18 = (arg0.w[14][22] * Val(2));
  Val x19 = (x17 - (x18 * arg0.w[14][7]));
  Val x20 = (arg0.w[14][23] + arg0.w[14][8]);
  Val x21 = (arg0.w[14][23] * Val(2));
  Val x22 = (x20 - (x21 * arg0.w[14][8]));
  Val x23 = (arg0.w[14][24] + arg0.w[14][9]);
  Val x24 = (arg0.w[14][24] * Val(2));
  Val x25 = (x23 - (x24 * arg0.w[14][9]));
  Val x26 = (arg0.w[14][25] + arg0.w[14][10]);
  Val x27 = (arg0.w[14][25] * Val(2));
  Val x28 = (x26 - (x27 * arg0.w[14][10]));
  Val x29 = (arg0.w[14][26] + arg0.w[14][11]);
  Val x30 = (arg0.w[14][26] * Val(2));
  Val x31 = (x29 - (x30 * arg0.w[14][11]));
  Val x32 = (arg0.w[14][27] + arg0.w[14][12]);
  Val x33 = (arg0.w[14][27] * Val(2));
  Val x34 = (x32 - (x33 * arg0.w[14][12]));
  Val x35 = (arg0.w[14][28] + arg0.w[14][13]);
  Val x36 = (arg0.w[14][28] * Val(2));
  Val x37 = (x35 - (x36 * arg0.w[14][13]));
  Val x38 = (arg0.w[14][29] + arg0.w[14][14]);
  Val x39 = (arg0.w[14][29] * Val(2));
  Val x40 = (x38 - (x39 * arg0.w[14][14]));
  Val x41 = (arg0.w[14][30] + arg0.w[14][15]);
  Val x42 = (arg0.w[14][30] * Val(2));
  Val x43 = (x41 - (x42 * arg0.w[14][15]));
  Val x44 = (arg0.w[14][31] + arg0.w[14][16]);
  Val x45 = (arg0.w[14][31] * Val(2));
  Val x46 = (x44 - (x45 * arg0.w[14][16]));
  Val x47 = (arg0.w[14][0] + arg0.w[14][17]);
  Val x48 = (arg0.w[14][0] * Val(2));
  Val x49 = (x47 - (x48 * arg0.w[14][17]));
  Val x50 = (arg0.w[14][1] + arg0.w[14][18]);
  Val x51 = (arg0.w[14][1] * Val(2));
  Val x52 = (x50 - (x51 * arg0.w[14][18]));
  Val x53 = (arg0.w[14][2] + arg0.w[14][19]);
  Val x54 = (arg0.w[14][2] * Val(2));
  Val x55 = (x53 - (x54 * arg0.w[14][19]));
  Val x56 = (arg0.w[14][3] + arg0.w[14][20]);
  Val x57 = (arg0.w[14][3] * Val(2));
  Val x58 = (x56 - (x57 * arg0.w[14][20]));
  Val x59 = (arg0.w[14][4] + arg0.w[14][21]);
  Val x60 = (arg0.w[14][4] * Val(2));
  Val x61 = (x59 - (x60 * arg0.w[14][21]));
  Val x62 = (arg0.w[14][5] + arg0.w[14][22]);
  Val x63 = (arg0.w[14][5] * Val(2));
  Val x64 = (x62 - (x63 * arg0.w[14][22]));
  Val x65 = (arg0.w[14][6] + arg0.w[14][23]);
  Val x66 = (arg0.w[14][6] * Val(2));
  Val x67 = (x65 - (x66 * arg0.w[14][23]));
  Val x68 = (arg0.w[14][7] + arg0.w[14][24]);
  Val x69 = (arg0.w[14][7] * Val(2));
  Val x70 = (x68 - (x69 * arg0.w[14][24]));
  Val x71 = (arg0.w[14][8] + arg0.w[14][25]);
  Val x72 = (arg0.w[14][8] * Val(2));
  Val x73 = (x71 - (x72 * arg0.w[14][25]));
  Val x74 = (arg0.w[14][9] + arg0.w[14][26]);
  Val x75 = (arg0.w[14][9] * Val(2));
  Val x76 = (x74 - (x75 * arg0.w[14][26]));
  Val x77 = (arg0.w[14][10] + arg0.w[14][27]);
  Val x78 = (arg0.w[14][10] * Val(2));
  Val x79 = (x77 - (x78 * arg0.w[14][27]));
  Val x80 = (arg0.w[14][11] + arg0.w[14][28]);
  Val x81 = (arg0.w[14][11] * Val(2));
  Val x82 = (x80 - (x81 * arg0.w[14][28]));
  Val x83 = (arg0.w[14][12] + arg0.w[14][29]);
  Val x84 = (arg0.w[14][12] * Val(2));
  Val x85 = (x83 - (x84 * arg0.w[14][29]));
  Val x86 = (arg0.w[14][13] + arg0.w[14][30]);
  Val x87 = (arg0.w[14][13] * Val(2));
  Val x88 = (x86 - (x87 * arg0.w[14][30]));
  Val x89 = (arg0.w[14][14] + arg0.w[14][31]);
  Val x90 = (arg0.w[14][14] * Val(2));
  Val x91 = (x89 - (x90 * arg0.w[14][31]));
  Val x92 = (arg0.w[14][7] + x7);
  Val x93 = (arg0.w[14][8] + x10);
  Val x94 = (arg0.w[14][9] + x13);
  Val x95 = (arg0.w[14][10] + x16);
  Val x96 = (arg0.w[14][11] + x19);
  Val x97 = (arg0.w[14][12] + x22);
  Val x98 = (arg0.w[14][13] + x25);
  Val x99 = (arg0.w[14][14] + x28);
  Val x100 = (arg0.w[14][15] + x31);
  Val x101 = (arg0.w[14][15] * Val(2));
  Val x102 = (arg0.w[14][16] + x34);
  Val x103 = (arg0.w[14][16] * Val(2));
  Val x104 = (arg0.w[14][17] + x37);
  Val x105 = (arg0.w[14][17] * Val(2));
  Val x106 = (arg0.w[14][18] + x40);
  Val x107 = (arg0.w[14][19] + x43);
  Val x108 = (arg0.w[14][20] + x46);
  Val x109 = (arg0.w[14][21] + x49);
  Val x110 = (arg0.w[14][22] + x52);
  Val x111 = (arg0.w[14][23] + x55);
  Val x112 = (arg0.w[14][24] + x58);
  Val x113 = (arg0.w[14][25] + x61);
  Val x114 = (arg0.w[14][26] + x64);
  Val x115 = (arg0.w[14][27] + x67);
  Val x116 = (arg0.w[14][28] + x70);
  Val x117 = (arg0.w[14][29] + x73);
  Val x118 = (arg0.w[14][30] + x76);
  Val x119 = (arg0.w[14][31] + x79);
  Val x120 = (arg0.w[14][0] + x82);
  Val x121 = (arg0.w[14][1] + x85);
  Val x122 = (arg0.w[14][2] + x88);
  Val x123 = (arg0.w[14][3] + x91);
  Val x124 = (arg0.w[14][4] + arg0.w[14][15]);
  Val x125 = (arg0.w[14][5] + arg0.w[14][16]);
  Val x126 = (arg0.w[14][6] + arg0.w[14][17]);
  // ComputeW(zirgen/circuit/keccak2/sha2.zir:60)
  Val x127 = (arg0.w[1][19] + arg0.w[1][10]);
  Val x128 = (arg0.w[1][19] * Val(2));
  Val x129 = (x127 - (x128 * arg0.w[1][10]));
  Val x130 = (arg0.w[1][20] + arg0.w[1][11]);
  Val x131 = (arg0.w[1][20] * Val(2));
  Val x132 = (x130 - (x131 * arg0.w[1][11]));
  Val x133 = (arg0.w[1][21] + arg0.w[1][12]);
  Val x134 = (arg0.w[1][21] * Val(2));
  Val x135 = (x133 - (x134 * arg0.w[1][12]));
  Val x136 = (arg0.w[1][22] + arg0.w[1][13]);
  Val x137 = (arg0.w[1][22] * Val(2));
  Val x138 = (x136 - (x137 * arg0.w[1][13]));
  Val x139 = (arg0.w[1][23] + arg0.w[1][14]);
  Val x140 = (arg0.w[1][23] * Val(2));
  Val x141 = (x139 - (x140 * arg0.w[1][14]));
  Val x142 = (arg0.w[1][24] + arg0.w[1][15]);
  Val x143 = (arg0.w[1][24] * Val(2));
  Val x144 = (x142 - (x143 * arg0.w[1][15]));
  Val x145 = (arg0.w[1][25] + arg0.w[1][16]);
  Val x146 = (arg0.w[1][25] * Val(2));
  Val x147 = (x145 - (x146 * arg0.w[1][16]));
  Val x148 = (arg0.w[1][26] + arg0.w[1][17]);
  Val x149 = (arg0.w[1][26] * Val(2));
  Val x150 = (x148 - (x149 * arg0.w[1][17]));
  Val x151 = (arg0.w[1][27] + arg0.w[1][18]);
  Val x152 = (arg0.w[1][27] * Val(2));
  Val x153 = (x151 - (x152 * arg0.w[1][18]));
  Val x154 = (arg0.w[1][28] + arg0.w[1][19]);
  Val x155 = (arg0.w[1][28] * Val(2));
  Val x156 = (x154 - (x155 * arg0.w[1][19]));
  Val x157 = (arg0.w[1][29] + arg0.w[1][20]);
  Val x158 = (arg0.w[1][29] * Val(2));
  Val x159 = (x157 - (x158 * arg0.w[1][20]));
  Val x160 = (arg0.w[1][30] + arg0.w[1][21]);
  Val x161 = (arg0.w[1][30] * Val(2));
  Val x162 = (x160 - (x161 * arg0.w[1][21]));
  Val x163 = (arg0.w[1][31] + arg0.w[1][22]);
  Val x164 = (arg0.w[1][31] * Val(2));
  Val x165 = (x163 - (x164 * arg0.w[1][22]));
  Val x166 = (arg0.w[1][0] + arg0.w[1][23]);
  Val x167 = (arg0.w[1][0] * Val(2));
  Val x168 = (x166 - (x167 * arg0.w[1][23]));
  Val x169 = (arg0.w[1][1] + arg0.w[1][24]);
  Val x170 = (arg0.w[1][1] * Val(2));
  Val x171 = (x169 - (x170 * arg0.w[1][24]));
  Val x172 = (arg0.w[1][2] + arg0.w[1][25]);
  Val x173 = (arg0.w[1][2] * Val(2));
  Val x174 = (x172 - (x173 * arg0.w[1][25]));
  Val x175 = (arg0.w[1][3] + arg0.w[1][26]);
  Val x176 = (arg0.w[1][3] * Val(2));
  Val x177 = (x175 - (x176 * arg0.w[1][26]));
  Val x178 = (arg0.w[1][4] + arg0.w[1][27]);
  Val x179 = (arg0.w[1][4] * Val(2));
  Val x180 = (x178 - (x179 * arg0.w[1][27]));
  Val x181 = (arg0.w[1][5] + arg0.w[1][28]);
  Val x182 = (arg0.w[1][5] * Val(2));
  Val x183 = (x181 - (x182 * arg0.w[1][28]));
  Val x184 = (arg0.w[1][6] + arg0.w[1][29]);
  Val x185 = (arg0.w[1][6] * Val(2));
  Val x186 = (x184 - (x185 * arg0.w[1][29]));
  Val x187 = (arg0.w[1][7] + arg0.w[1][30]);
  Val x188 = (arg0.w[1][7] * Val(2));
  Val x189 = (x187 - (x188 * arg0.w[1][30]));
  Val x190 = (arg0.w[1][8] + arg0.w[1][31]);
  Val x191 = (arg0.w[1][8] * Val(2));
  Val x192 = (x190 - (x191 * arg0.w[1][31]));
  Val x193 = (arg0.w[1][17] + x129);
  Val x194 = (arg0.w[1][17] * Val(2));
  Val x195 = (arg0.w[1][18] + x132);
  Val x196 = (arg0.w[1][18] * Val(2));
  Val x197 = (arg0.w[1][19] + x135);
  Val x198 = (arg0.w[1][20] + x138);
  Val x199 = (arg0.w[1][21] + x141);
  Val x200 = (arg0.w[1][22] + x144);
  Val x201 = (arg0.w[1][23] + x147);
  Val x202 = (arg0.w[1][24] + x150);
  Val x203 = (arg0.w[1][25] + x153);
  Val x204 = (arg0.w[1][26] + x156);
  Val x205 = (arg0.w[1][27] + x159);
  Val x206 = (arg0.w[1][28] + x162);
  Val x207 = (arg0.w[1][29] + x165);
  Val x208 = (arg0.w[1][30] + x168);
  Val x209 = (arg0.w[1][31] + x171);
  Val x210 = (arg0.w[1][0] + x174);
  Val x211 = (arg0.w[1][1] + x177);
  Val x212 = (arg0.w[1][2] + x180);
  Val x213 = (arg0.w[1][3] + x183);
  Val x214 = (arg0.w[1][4] + x186);
  Val x215 = (arg0.w[1][5] + x189);
  Val x216 = (arg0.w[1][6] + x192);
  Val x217 = (arg0.w[1][7] + arg0.w[1][9]);
  Val x218 = (arg0.w[1][8] + arg0.w[1][10]);
  Val x219 = (arg0.w[1][9] + arg0.w[1][11]);
  Val x220 = (arg0.w[1][9] * Val(2));
  Val x221 = (arg0.w[1][10] + arg0.w[1][12]);
  Val x222 = (arg0.w[1][10] * Val(2));
  Val x223 = (arg0.w[1][11] + arg0.w[1][13]);
  Val x224 = (arg0.w[1][11] * Val(2));
  Val x225 = (arg0.w[1][12] + arg0.w[1][14]);
  Val x226 = (arg0.w[1][12] * Val(2));
  Val x227 = (arg0.w[1][13] + arg0.w[1][15]);
  Val x228 = (arg0.w[1][13] * Val(2));
  Val x229 = (arg0.w[1][14] + arg0.w[1][16]);
  Val x230 = (arg0.w[1][14] * Val(2));
  Val x231 = (arg0.w[1][15] + arg0.w[1][17]);
  Val x232 = (arg0.w[1][15] * Val(2));
  Val x233 = (arg0.w[1][16] + arg0.w[1][18]);
  Val x234 = (arg0.w[1][16] * Val(2));
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  // ComputeW(zirgen/circuit/keccak2/sha2.zir:61)
  Val x235 = ((x92 - (x69 * x7)) + ((x93 - (x72 * x10)) * Val(2)));
  Val x236 = ((x235 + ((x94 - (x75 * x13)) * Val(4))) + ((x95 - (x78 * x16)) * Val(8)));
  Val x237 = ((x236 + ((x96 - (x81 * x19)) * Val(16))) + ((x97 - (x84 * x22)) * Val(32)));
  Val x238 = ((x237 + ((x98 - (x87 * x25)) * Val(64))) + ((x99 - (x90 * x28)) * Val(128)));
  Val x239 = ((x238 + ((x100 - (x101 * x31)) * Val(256))) + ((x102 - (x103 * x34)) * Val(512)));
  Val x240 = ((x239 + ((x104 - (x105 * x37)) * Val(1024))) + ((x106 - (x6 * x40)) * Val(2048)));
  Val x241 = ((x240 + ((x107 - (x9 * x43)) * Val(4096))) + ((x108 - (x12 * x46)) * Val(8192)));
  Val x242 = ((x241 + ((x109 - (x15 * x49)) * Val(16384))) + ((x110 - (x18 * x52)) * Val(32768)));
  Val x243 = ((x111 - (x21 * x55)) + ((x112 - (x24 * x58)) * Val(2)));
  Val x244 = ((x243 + ((x113 - (x27 * x61)) * Val(4))) + ((x114 - (x30 * x64)) * Val(8)));
  Val x245 = ((x244 + ((x115 - (x33 * x67)) * Val(16))) + ((x116 - (x36 * x70)) * Val(32)));
  Val x246 = ((x245 + ((x117 - (x39 * x73)) * Val(64))) + ((x118 - (x42 * x76)) * Val(128)));
  Val x247 = ((x246 + ((x119 - (x45 * x79)) * Val(256))) + ((x120 - (x48 * x82)) * Val(512)));
  Val x248 = ((x247 + ((x121 - (x51 * x85)) * Val(1024))) + ((x122 - (x54 * x88)) * Val(2048)));
  Val x249 =
      ((x248 + ((x123 - (x57 * x91)) * Val(4096))) + ((x124 - (x60 * arg0.w[14][15])) * Val(8192)));
  Val x250 = ((x249 + ((x125 - (x63 * arg0.w[14][16])) * Val(16384))) +
              ((x126 - (x66 * arg0.w[14][17])) * Val(32768)));
  Val x251 = ((x193 - (x194 * x129)) + ((x195 - (x196 * x132)) * Val(2)));
  Val x252 = ((x251 + ((x197 - (x128 * x135)) * Val(4))) + ((x198 - (x131 * x138)) * Val(8)));
  Val x253 = ((x252 + ((x199 - (x134 * x141)) * Val(16))) + ((x200 - (x137 * x144)) * Val(32)));
  Val x254 = ((x253 + ((x201 - (x140 * x147)) * Val(64))) + ((x202 - (x143 * x150)) * Val(128)));
  Val x255 = ((x254 + ((x203 - (x146 * x153)) * Val(256))) + ((x204 - (x149 * x156)) * Val(512)));
  Val x256 = ((x255 + ((x205 - (x152 * x159)) * Val(1024))) + ((x206 - (x155 * x162)) * Val(2048)));
  Val x257 = ((x256 + ((x207 - (x158 * x165)) * Val(4096))) + ((x208 - (x161 * x168)) * Val(8192)));
  Val x258 =
      ((x257 + ((x209 - (x164 * x171)) * Val(16384))) + ((x210 - (x167 * x174)) * Val(32768)));
  Val x259 = ((x211 - (x170 * x177)) + ((x212 - (x173 * x180)) * Val(2)));
  Val x260 = ((x259 + ((x213 - (x176 * x183)) * Val(4))) + ((x214 - (x179 * x186)) * Val(8)));
  Val x261 = ((x260 + ((x215 - (x182 * x189)) * Val(16))) + ((x216 - (x185 * x192)) * Val(32)));
  Val x262 = ((x261 + ((x217 - (x188 * arg0.w[1][9])) * Val(64))) +
              ((x218 - (x191 * arg0.w[1][10])) * Val(128)));
  Val x263 = ((x262 + ((x219 - (x220 * arg0.w[1][11])) * Val(256))) +
              ((x221 - (x222 * arg0.w[1][12])) * Val(512)));
  Val x264 = ((x263 + ((x223 - (x224 * arg0.w[1][13])) * Val(1024))) +
              ((x225 - (x226 * arg0.w[1][14])) * Val(2048)));
  Val x265 = ((x264 + ((x227 - (x228 * arg0.w[1][15])) * Val(4096))) +
              ((x229 - (x230 * arg0.w[1][16])) * Val(8192)));
  Val x266 = ((x265 + ((x231 - (x232 * arg0.w[1][17])) * Val(16384))) +
              ((x233 - (x234 * arg0.w[1][18])) * Val(32768)));
  Val x267 = (arg0.w[15][1] * Val(2));
  Val x268 = (arg0.w[15][2] * Val(4));
  Val x269 = (arg0.w[15][3] * Val(8));
  Val x270 = (arg0.w[15][4] * Val(16));
  Val x271 = (arg0.w[15][5] * Val(32));
  Val x272 = (arg0.w[15][6] * Val(64));
  Val x273 = (arg0.w[15][7] * Val(128));
  Val x274 = (arg0.w[15][8] * Val(256));
  Val x275 = (arg0.w[15][9] * Val(512));
  Val x276 = (arg0.w[15][10] * Val(1024));
  Val x277 = (arg0.w[15][11] * Val(2048));
  Val x278 = (arg0.w[15][12] * Val(4096));
  Val x279 = (arg0.w[15][13] * Val(8192));
  Val x280 = (arg0.w[15][14] * Val(16384));
  Val x281 = (arg0.w[15][15] * Val(32768));
  Val x282 = (arg0.w[15][0] + x267);
  Val x283 = (((x282 + x268) + x269) + x270);
  Val x284 = (((x283 + x271) + x272) + x273);
  Val x285 = (((x284 + x274) + x275) + x276);
  Val x286 = (((x285 + x277) + x278) + x279);
  Val x287 = (arg0.w[15][17] * Val(2));
  Val x288 = (arg0.w[15][18] * Val(4));
  Val x289 = (arg0.w[15][19] * Val(8));
  Val x290 = (arg0.w[15][20] * Val(16));
  Val x291 = (arg0.w[15][21] * Val(32));
  Val x292 = (arg0.w[15][22] * Val(64));
  Val x293 = (arg0.w[15][23] * Val(128));
  Val x294 = (arg0.w[15][24] * Val(256));
  Val x295 = (arg0.w[15][25] * Val(512));
  Val x296 = (arg0.w[15][26] * Val(1024));
  Val x297 = (arg0.w[15][27] * Val(2048));
  Val x298 = (arg0.w[15][28] * Val(4096));
  Val x299 = (arg0.w[15][29] * Val(8192));
  Val x300 = (arg0.w[15][30] * Val(16384));
  Val x301 = (arg0.w[15][31] * Val(32768));
  Val x302 = (arg0.w[15][16] + x287);
  Val x303 = (((x302 + x288) + x289) + x290);
  Val x304 = (((x303 + x291) + x292) + x293);
  Val x305 = (((x304 + x294) + x295) + x296);
  Val x306 = (((x305 + x297) + x298) + x299);
  Val x307 = (arg0.w[6][1] * Val(2));
  Val x308 = (arg0.w[6][2] * Val(4));
  Val x309 = (arg0.w[6][3] * Val(8));
  Val x310 = (arg0.w[6][4] * Val(16));
  Val x311 = (arg0.w[6][5] * Val(32));
  Val x312 = (arg0.w[6][6] * Val(64));
  Val x313 = (arg0.w[6][7] * Val(128));
  Val x314 = (arg0.w[6][8] * Val(256));
  Val x315 = (arg0.w[6][9] * Val(512));
  Val x316 = (arg0.w[6][10] * Val(1024));
  Val x317 = (arg0.w[6][11] * Val(2048));
  Val x318 = (arg0.w[6][12] * Val(4096));
  Val x319 = (arg0.w[6][13] * Val(8192));
  Val x320 = (arg0.w[6][14] * Val(16384));
  Val x321 = (arg0.w[6][15] * Val(32768));
  Val x322 = (arg0.w[6][0] + x307);
  Val x323 = (((x322 + x308) + x309) + x310);
  Val x324 = (((x323 + x311) + x312) + x313);
  Val x325 = (((x324 + x314) + x315) + x316);
  Val x326 = (((x325 + x317) + x318) + x319);
  Val x327 = (arg0.w[6][17] * Val(2));
  Val x328 = (arg0.w[6][18] * Val(4));
  Val x329 = (arg0.w[6][19] * Val(8));
  Val x330 = (arg0.w[6][20] * Val(16));
  Val x331 = (arg0.w[6][21] * Val(32));
  Val x332 = (arg0.w[6][22] * Val(64));
  Val x333 = (arg0.w[6][23] * Val(128));
  Val x334 = (arg0.w[6][24] * Val(256));
  Val x335 = (arg0.w[6][25] * Val(512));
  Val x336 = (arg0.w[6][26] * Val(1024));
  Val x337 = (arg0.w[6][27] * Val(2048));
  Val x338 = (arg0.w[6][28] * Val(4096));
  Val x339 = (arg0.w[6][29] * Val(8192));
  Val x340 = (arg0.w[6][30] * Val(16384));
  Val x341 = (arg0.w[6][31] * Val(32768));
  Val x342 = (arg0.w[6][16] + x327);
  Val x343 = (((x342 + x328) + x329) + x330);
  Val x344 = (((x343 + x331) + x332) + x333);
  Val x345 = (((x344 + x334) + x335) + x336);
  Val x346 = (((x345 + x337) + x338) + x339);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x347 = (((x286 + x280) + x281) + ((x326 + x320) + x321));
  Val x348 = (((x306 + x300) + x301) + ((x346 + x340) + x341));
  // DoShaStep(zirgen/circuit/keccak2/top.zir:138)
  Val x349 = (Val(1) - arg2);
  Val2Array x350 = Val2Array{((arg2 * arg3[0]) + (x349 * (x242 + (x258 + x347)))),
                             ((arg2 * arg3[1]) + (x349 * (x250 + (x266 + x348))))};
  // DoShaStep(zirgen/circuit/keccak2/top.zir:140)
  UnpackReg_32__16_Struct x351 = exec_CarryAndExpand(ctx, x350, LAYOUT_LOOKUP(layout4, w));
  // BitXor(zirgen/circuit/keccak2/bits.zir:32)
  // XorU32(zirgen/circuit/keccak2/sha2.zir:10)
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:72)
  // DoShaStep(zirgen/circuit/keccak2/top.zir:141)
  Val x352 = (arg0.a[0][13] + arg0.a[0][22]);
  Val x353 = (arg0.a[0][13] * Val(2));
  Val x354 = (x352 - (x353 * arg0.a[0][22]));
  Val x355 = (arg0.a[0][14] + arg0.a[0][23]);
  Val x356 = (arg0.a[0][14] * Val(2));
  Val x357 = (x355 - (x356 * arg0.a[0][23]));
  Val x358 = (arg0.a[0][15] + arg0.a[0][24]);
  Val x359 = (arg0.a[0][15] * Val(2));
  Val x360 = (x358 - (x359 * arg0.a[0][24]));
  Val x361 = (arg0.a[0][16] + arg0.a[0][25]);
  Val x362 = (arg0.a[0][16] * Val(2));
  Val x363 = (x361 - (x362 * arg0.a[0][25]));
  Val x364 = (arg0.a[0][17] + arg0.a[0][26]);
  Val x365 = (arg0.a[0][17] * Val(2));
  Val x366 = (x364 - (x365 * arg0.a[0][26]));
  Val x367 = (arg0.a[0][18] + arg0.a[0][27]);
  Val x368 = (arg0.a[0][18] * Val(2));
  Val x369 = (x367 - (x368 * arg0.a[0][27]));
  Val x370 = (arg0.a[0][19] + arg0.a[0][28]);
  Val x371 = (arg0.a[0][19] * Val(2));
  Val x372 = (x370 - (x371 * arg0.a[0][28]));
  Val x373 = (arg0.a[0][20] + arg0.a[0][29]);
  Val x374 = (arg0.a[0][20] * Val(2));
  Val x375 = (x373 - (x374 * arg0.a[0][29]));
  Val x376 = (arg0.a[0][21] + arg0.a[0][30]);
  Val x377 = (arg0.a[0][21] * Val(2));
  Val x378 = (x376 - (x377 * arg0.a[0][30]));
  Val x379 = (arg0.a[0][22] + arg0.a[0][31]);
  Val x380 = (arg0.a[0][22] * Val(2));
  Val x381 = (x379 - (x380 * arg0.a[0][31]));
  Val x382 = (arg0.a[0][23] + arg0.a[0][0]);
  Val x383 = (arg0.a[0][23] * Val(2));
  Val x384 = (x382 - (x383 * arg0.a[0][0]));
  Val x385 = (arg0.a[0][24] + arg0.a[0][1]);
  Val x386 = (arg0.a[0][24] * Val(2));
  Val x387 = (x385 - (x386 * arg0.a[0][1]));
  Val x388 = (arg0.a[0][25] + arg0.a[0][2]);
  Val x389 = (arg0.a[0][25] * Val(2));
  Val x390 = (x388 - (x389 * arg0.a[0][2]));
  Val x391 = (arg0.a[0][26] + arg0.a[0][3]);
  Val x392 = (arg0.a[0][26] * Val(2));
  Val x393 = (x391 - (x392 * arg0.a[0][3]));
  Val x394 = (arg0.a[0][27] + arg0.a[0][4]);
  Val x395 = (arg0.a[0][27] * Val(2));
  Val x396 = (x394 - (x395 * arg0.a[0][4]));
  Val x397 = (arg0.a[0][28] + arg0.a[0][5]);
  Val x398 = (arg0.a[0][28] * Val(2));
  Val x399 = (x397 - (x398 * arg0.a[0][5]));
  Val x400 = (arg0.a[0][29] + arg0.a[0][6]);
  Val x401 = (arg0.a[0][29] * Val(2));
  Val x402 = (x400 - (x401 * arg0.a[0][6]));
  Val x403 = (arg0.a[0][30] + arg0.a[0][7]);
  Val x404 = (arg0.a[0][30] * Val(2));
  Val x405 = (x403 - (x404 * arg0.a[0][7]));
  Val x406 = (arg0.a[0][31] + arg0.a[0][8]);
  Val x407 = (arg0.a[0][31] * Val(2));
  Val x408 = (x406 - (x407 * arg0.a[0][8]));
  Val x409 = (arg0.a[0][0] + arg0.a[0][9]);
  Val x410 = (arg0.a[0][0] * Val(2));
  Val x411 = (x409 - (x410 * arg0.a[0][9]));
  Val x412 = (arg0.a[0][1] + arg0.a[0][10]);
  Val x413 = (arg0.a[0][1] * Val(2));
  Val x414 = (x412 - (x413 * arg0.a[0][10]));
  Val x415 = (arg0.a[0][2] + arg0.a[0][11]);
  Val x416 = (arg0.a[0][2] * Val(2));
  Val x417 = (x415 - (x416 * arg0.a[0][11]));
  Val x418 = (arg0.a[0][3] + arg0.a[0][12]);
  Val x419 = (arg0.a[0][3] * Val(2));
  Val x420 = (x418 - (x419 * arg0.a[0][12]));
  Val x421 = (arg0.a[0][4] + arg0.a[0][13]);
  Val x422 = (arg0.a[0][4] * Val(2));
  Val x423 = (x421 - (x422 * arg0.a[0][13]));
  Val x424 = (arg0.a[0][5] + arg0.a[0][14]);
  Val x425 = (arg0.a[0][5] * Val(2));
  Val x426 = (x424 - (x425 * arg0.a[0][14]));
  Val x427 = (arg0.a[0][6] + arg0.a[0][15]);
  Val x428 = (arg0.a[0][6] * Val(2));
  Val x429 = (x427 - (x428 * arg0.a[0][15]));
  Val x430 = (arg0.a[0][7] + arg0.a[0][16]);
  Val x431 = (arg0.a[0][7] * Val(2));
  Val x432 = (x430 - (x431 * arg0.a[0][16]));
  Val x433 = (arg0.a[0][8] + arg0.a[0][17]);
  Val x434 = (arg0.a[0][8] * Val(2));
  Val x435 = (x433 - (x434 * arg0.a[0][17]));
  Val x436 = (arg0.a[0][9] + arg0.a[0][18]);
  Val x437 = (arg0.a[0][9] * Val(2));
  Val x438 = (x436 - (x437 * arg0.a[0][18]));
  Val x439 = (arg0.a[0][10] + arg0.a[0][19]);
  Val x440 = (arg0.a[0][10] * Val(2));
  Val x441 = (x439 - (x440 * arg0.a[0][19]));
  Val x442 = (arg0.a[0][11] + arg0.a[0][20]);
  Val x443 = (arg0.a[0][11] * Val(2));
  Val x444 = (x442 - (x443 * arg0.a[0][20]));
  Val x445 = (arg0.a[0][12] + arg0.a[0][21]);
  Val x446 = (arg0.a[0][12] * Val(2));
  Val x447 = (x445 - (x446 * arg0.a[0][21]));
  Val x448 = (arg0.a[0][2] + x354);
  Val x449 = (arg0.a[0][3] + x357);
  Val x450 = (arg0.a[0][4] + x360);
  Val x451 = (arg0.a[0][5] + x363);
  Val x452 = (arg0.a[0][6] + x366);
  Val x453 = (arg0.a[0][7] + x369);
  Val x454 = (arg0.a[0][8] + x372);
  Val x455 = (arg0.a[0][9] + x375);
  Val x456 = (arg0.a[0][10] + x378);
  Val x457 = (arg0.a[0][11] + x381);
  Val x458 = (arg0.a[0][12] + x384);
  Val x459 = (arg0.a[0][13] + x387);
  Val x460 = (arg0.a[0][14] + x390);
  Val x461 = (arg0.a[0][15] + x393);
  Val x462 = (arg0.a[0][16] + x396);
  Val x463 = (arg0.a[0][17] + x399);
  Val x464 = (arg0.a[0][18] + x402);
  Val x465 = (arg0.a[0][19] + x405);
  Val x466 = (arg0.a[0][20] + x408);
  Val x467 = (arg0.a[0][21] + x411);
  Val x468 = (arg0.a[0][22] + x414);
  Val x469 = (arg0.a[0][23] + x417);
  Val x470 = (arg0.a[0][24] + x420);
  Val x471 = (arg0.a[0][25] + x423);
  Val x472 = (arg0.a[0][26] + x426);
  Val x473 = (arg0.a[0][27] + x429);
  Val x474 = (arg0.a[0][28] + x432);
  Val x475 = (arg0.a[0][29] + x435);
  Val x476 = (arg0.a[0][30] + x438);
  Val x477 = (arg0.a[0][31] + x441);
  Val x478 = (arg0.a[0][0] + x444);
  Val x479 = (arg0.a[0][1] + x447);
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:73)
  Val x480 = (arg0.e[0][11] + arg0.e[0][25]);
  Val x481 = (arg0.e[0][11] * Val(2));
  Val x482 = (x480 - (x481 * arg0.e[0][25]));
  Val x483 = (arg0.e[0][12] + arg0.e[0][26]);
  Val x484 = (arg0.e[0][12] * Val(2));
  Val x485 = (x483 - (x484 * arg0.e[0][26]));
  Val x486 = (arg0.e[0][13] + arg0.e[0][27]);
  Val x487 = (arg0.e[0][13] * Val(2));
  Val x488 = (x486 - (x487 * arg0.e[0][27]));
  Val x489 = (arg0.e[0][14] + arg0.e[0][28]);
  Val x490 = (arg0.e[0][14] * Val(2));
  Val x491 = (x489 - (x490 * arg0.e[0][28]));
  Val x492 = (arg0.e[0][15] + arg0.e[0][29]);
  Val x493 = (arg0.e[0][15] * Val(2));
  Val x494 = (x492 - (x493 * arg0.e[0][29]));
  Val x495 = (arg0.e[0][16] + arg0.e[0][30]);
  Val x496 = (arg0.e[0][16] * Val(2));
  Val x497 = (x495 - (x496 * arg0.e[0][30]));
  Val x498 = (arg0.e[0][17] + arg0.e[0][31]);
  Val x499 = (arg0.e[0][17] * Val(2));
  Val x500 = (x498 - (x499 * arg0.e[0][31]));
  Val x501 = (arg0.e[0][18] + arg0.e[0][0]);
  Val x502 = (arg0.e[0][18] * Val(2));
  Val x503 = (x501 - (x502 * arg0.e[0][0]));
  Val x504 = (arg0.e[0][19] + arg0.e[0][1]);
  Val x505 = (arg0.e[0][19] * Val(2));
  Val x506 = (x504 - (x505 * arg0.e[0][1]));
  Val x507 = (arg0.e[0][20] + arg0.e[0][2]);
  Val x508 = (arg0.e[0][20] * Val(2));
  Val x509 = (x507 - (x508 * arg0.e[0][2]));
  Val x510 = (arg0.e[0][21] + arg0.e[0][3]);
  Val x511 = (arg0.e[0][21] * Val(2));
  Val x512 = (x510 - (x511 * arg0.e[0][3]));
  Val x513 = (arg0.e[0][22] + arg0.e[0][4]);
  Val x514 = (arg0.e[0][22] * Val(2));
  Val x515 = (x513 - (x514 * arg0.e[0][4]));
  Val x516 = (arg0.e[0][23] + arg0.e[0][5]);
  Val x517 = (arg0.e[0][23] * Val(2));
  Val x518 = (x516 - (x517 * arg0.e[0][5]));
  Val x519 = (arg0.e[0][24] + arg0.e[0][6]);
  Val x520 = (arg0.e[0][24] * Val(2));
  Val x521 = (x519 - (x520 * arg0.e[0][6]));
  Val x522 = (arg0.e[0][25] + arg0.e[0][7]);
  Val x523 = (arg0.e[0][25] * Val(2));
  Val x524 = (x522 - (x523 * arg0.e[0][7]));
  Val x525 = (arg0.e[0][26] + arg0.e[0][8]);
  Val x526 = (arg0.e[0][26] * Val(2));
  Val x527 = (x525 - (x526 * arg0.e[0][8]));
  Val x528 = (arg0.e[0][27] + arg0.e[0][9]);
  Val x529 = (arg0.e[0][27] * Val(2));
  Val x530 = (x528 - (x529 * arg0.e[0][9]));
  Val x531 = (arg0.e[0][28] + arg0.e[0][10]);
  Val x532 = (arg0.e[0][28] * Val(2));
  Val x533 = (x531 - (x532 * arg0.e[0][10]));
  Val x534 = (arg0.e[0][29] + arg0.e[0][11]);
  Val x535 = (arg0.e[0][29] * Val(2));
  Val x536 = (x534 - (x535 * arg0.e[0][11]));
  Val x537 = (arg0.e[0][30] + arg0.e[0][12]);
  Val x538 = (arg0.e[0][30] * Val(2));
  Val x539 = (x537 - (x538 * arg0.e[0][12]));
  Val x540 = (arg0.e[0][31] + arg0.e[0][13]);
  Val x541 = (arg0.e[0][31] * Val(2));
  Val x542 = (x540 - (x541 * arg0.e[0][13]));
  Val x543 = (arg0.e[0][0] + arg0.e[0][14]);
  Val x544 = (arg0.e[0][0] * Val(2));
  Val x545 = (x543 - (x544 * arg0.e[0][14]));
  Val x546 = (arg0.e[0][1] + arg0.e[0][15]);
  Val x547 = (arg0.e[0][1] * Val(2));
  Val x548 = (x546 - (x547 * arg0.e[0][15]));
  Val x549 = (arg0.e[0][2] + arg0.e[0][16]);
  Val x550 = (arg0.e[0][2] * Val(2));
  Val x551 = (x549 - (x550 * arg0.e[0][16]));
  Val x552 = (arg0.e[0][3] + arg0.e[0][17]);
  Val x553 = (arg0.e[0][3] * Val(2));
  Val x554 = (x552 - (x553 * arg0.e[0][17]));
  Val x555 = (arg0.e[0][4] + arg0.e[0][18]);
  Val x556 = (arg0.e[0][4] * Val(2));
  Val x557 = (x555 - (x556 * arg0.e[0][18]));
  Val x558 = (arg0.e[0][5] + arg0.e[0][19]);
  Val x559 = (arg0.e[0][5] * Val(2));
  Val x560 = (x558 - (x559 * arg0.e[0][19]));
  Val x561 = (arg0.e[0][6] + arg0.e[0][20]);
  Val x562 = (arg0.e[0][6] * Val(2));
  Val x563 = (x561 - (x562 * arg0.e[0][20]));
  Val x564 = (arg0.e[0][7] + arg0.e[0][21]);
  Val x565 = (arg0.e[0][7] * Val(2));
  Val x566 = (x564 - (x565 * arg0.e[0][21]));
  Val x567 = (arg0.e[0][8] + arg0.e[0][22]);
  Val x568 = (arg0.e[0][8] * Val(2));
  Val x569 = (x567 - (x568 * arg0.e[0][22]));
  Val x570 = (arg0.e[0][9] + arg0.e[0][23]);
  Val x571 = (arg0.e[0][9] * Val(2));
  Val x572 = (x570 - (x571 * arg0.e[0][23]));
  Val x573 = (arg0.e[0][10] + arg0.e[0][24]);
  Val x574 = (arg0.e[0][10] * Val(2));
  Val x575 = (x573 - (x574 * arg0.e[0][24]));
  Val x576 = (arg0.e[0][6] + x482);
  Val x577 = (arg0.e[0][7] + x485);
  Val x578 = (arg0.e[0][8] + x488);
  Val x579 = (arg0.e[0][9] + x491);
  Val x580 = (arg0.e[0][10] + x494);
  Val x581 = (arg0.e[0][11] + x497);
  Val x582 = (arg0.e[0][12] + x500);
  Val x583 = (arg0.e[0][13] + x503);
  Val x584 = (arg0.e[0][14] + x506);
  Val x585 = (arg0.e[0][15] + x509);
  Val x586 = (arg0.e[0][16] + x512);
  Val x587 = (arg0.e[0][17] + x515);
  Val x588 = (arg0.e[0][18] + x518);
  Val x589 = (arg0.e[0][19] + x521);
  Val x590 = (arg0.e[0][20] + x524);
  Val x591 = (arg0.e[0][21] + x527);
  Val x592 = (arg0.e[0][22] + x530);
  Val x593 = (arg0.e[0][23] + x533);
  Val x594 = (arg0.e[0][24] + x536);
  Val x595 = (arg0.e[0][25] + x539);
  Val x596 = (arg0.e[0][26] + x542);
  Val x597 = (arg0.e[0][27] + x545);
  Val x598 = (arg0.e[0][28] + x548);
  Val x599 = (arg0.e[0][29] + x551);
  Val x600 = (arg0.e[0][30] + x554);
  Val x601 = (arg0.e[0][31] + x557);
  Val x602 = (arg0.e[0][0] + x560);
  Val x603 = (arg0.e[0][1] + x563);
  Val x604 = (arg0.e[0][2] + x566);
  Val x605 = (arg0.e[0][3] + x569);
  Val x606 = (arg0.e[0][4] + x572);
  Val x607 = (arg0.e[0][5] + x575);
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:74)
  Val x608 = (x351._super[1]._super._super * Val(2));
  Val x609 = (x351._super[2]._super._super * Val(4));
  Val x610 = (x351._super[3]._super._super * Val(8));
  Val x611 = (x351._super[4]._super._super * Val(16));
  Val x612 = (x351._super[5]._super._super * Val(32));
  Val x613 = (x351._super[6]._super._super * Val(64));
  Val x614 = (x351._super[7]._super._super * Val(128));
  Val x615 = (x351._super[8]._super._super * Val(256));
  Val x616 = (x351._super[9]._super._super * Val(512));
  Val x617 = (x351._super[10]._super._super * Val(1024));
  Val x618 = (x351._super[11]._super._super * Val(2048));
  Val x619 = (x351._super[12]._super._super * Val(4096));
  Val x620 = (x351._super[13]._super._super * Val(8192));
  Val x621 = (x351._super[14]._super._super * Val(16384));
  Val x622 = (x351._super[15]._super._super * Val(32768));
  Val x623 = (x351._super[0]._super._super + x608);
  Val x624 = (((x623 + x609) + x610) + x611);
  Val x625 = (((x624 + x612) + x613) + x614);
  Val x626 = (((x625 + x615) + x616) + x617);
  Val x627 = (((x626 + x618) + x619) + x620);
  Val x628 = (x351._super[17]._super._super * Val(2));
  Val x629 = (x351._super[18]._super._super * Val(4));
  Val x630 = (x351._super[19]._super._super * Val(8));
  Val x631 = (x351._super[20]._super._super * Val(16));
  Val x632 = (x351._super[21]._super._super * Val(32));
  Val x633 = (x351._super[22]._super._super * Val(64));
  Val x634 = (x351._super[23]._super._super * Val(128));
  Val x635 = (x351._super[24]._super._super * Val(256));
  Val x636 = (x351._super[25]._super._super * Val(512));
  Val x637 = (x351._super[26]._super._super * Val(1024));
  Val x638 = (x351._super[27]._super._super * Val(2048));
  Val x639 = (x351._super[28]._super._super * Val(4096));
  Val x640 = (x351._super[29]._super._super * Val(8192));
  Val x641 = (x351._super[30]._super._super * Val(16384));
  Val x642 = (x351._super[31]._super._super * Val(32768));
  Val x643 = (x351._super[16]._super._super + x628);
  Val x644 = (((x643 + x629) + x630) + x631);
  Val x645 = (((x644 + x632) + x633) + x634);
  Val x646 = (((x645 + x635) + x636) + x637);
  Val x647 = (((x646 + x638) + x639) + x640);
  Val x648 = (arg0.e[3][1] * Val(2));
  Val x649 = (arg0.e[3][2] * Val(4));
  Val x650 = (arg0.e[3][3] * Val(8));
  Val x651 = (arg0.e[3][4] * Val(16));
  Val x652 = (arg0.e[3][5] * Val(32));
  Val x653 = (arg0.e[3][6] * Val(64));
  Val x654 = (arg0.e[3][7] * Val(128));
  Val x655 = (arg0.e[3][8] * Val(256));
  Val x656 = (arg0.e[3][9] * Val(512));
  Val x657 = (arg0.e[3][10] * Val(1024));
  Val x658 = (arg0.e[3][11] * Val(2048));
  Val x659 = (arg0.e[3][12] * Val(4096));
  Val x660 = (arg0.e[3][13] * Val(8192));
  Val x661 = (arg0.e[3][14] * Val(16384));
  Val x662 = (arg0.e[3][15] * Val(32768));
  Val x663 = (arg0.e[3][0] + x648);
  Val x664 = (((x663 + x649) + x650) + x651);
  Val x665 = (((x664 + x652) + x653) + x654);
  Val x666 = (((x665 + x655) + x656) + x657);
  Val x667 = (((x666 + x658) + x659) + x660);
  Val x668 = (arg0.e[3][17] * Val(2));
  Val x669 = (arg0.e[3][18] * Val(4));
  Val x670 = (arg0.e[3][19] * Val(8));
  Val x671 = (arg0.e[3][20] * Val(16));
  Val x672 = (arg0.e[3][21] * Val(32));
  Val x673 = (arg0.e[3][22] * Val(64));
  Val x674 = (arg0.e[3][23] * Val(128));
  Val x675 = (arg0.e[3][24] * Val(256));
  Val x676 = (arg0.e[3][25] * Val(512));
  Val x677 = (arg0.e[3][26] * Val(1024));
  Val x678 = (arg0.e[3][27] * Val(2048));
  Val x679 = (arg0.e[3][28] * Val(4096));
  Val x680 = (arg0.e[3][29] * Val(8192));
  Val x681 = (arg0.e[3][30] * Val(16384));
  Val x682 = (arg0.e[3][31] * Val(32768));
  Val x683 = (arg0.e[3][16] + x668);
  Val x684 = (((x683 + x669) + x670) + x671);
  Val x685 = (((x684 + x672) + x673) + x674);
  Val x686 = (((x685 + x675) + x676) + x677);
  Val x687 = (((x686 + x678) + x679) + x680);
  // ChU32(zirgen/circuit/keccak2/sha2.zir:25)
  Val x688 = (arg0.e[0][0] * arg0.e[1][0]);
  Val x689 = (Val(1) - arg0.e[0][0]);
  Val x690 = (arg0.e[0][1] * arg0.e[1][1]);
  Val x691 = (Val(1) - arg0.e[0][1]);
  Val x692 = (arg0.e[0][2] * arg0.e[1][2]);
  Val x693 = (Val(1) - arg0.e[0][2]);
  Val x694 = (arg0.e[0][3] * arg0.e[1][3]);
  Val x695 = (Val(1) - arg0.e[0][3]);
  Val x696 = (arg0.e[0][4] * arg0.e[1][4]);
  Val x697 = (Val(1) - arg0.e[0][4]);
  Val x698 = (arg0.e[0][5] * arg0.e[1][5]);
  Val x699 = (Val(1) - arg0.e[0][5]);
  Val x700 = (arg0.e[0][6] * arg0.e[1][6]);
  Val x701 = (Val(1) - arg0.e[0][6]);
  Val x702 = (arg0.e[0][7] * arg0.e[1][7]);
  Val x703 = (Val(1) - arg0.e[0][7]);
  Val x704 = (arg0.e[0][8] * arg0.e[1][8]);
  Val x705 = (Val(1) - arg0.e[0][8]);
  Val x706 = (arg0.e[0][9] * arg0.e[1][9]);
  Val x707 = (Val(1) - arg0.e[0][9]);
  Val x708 = (arg0.e[0][10] * arg0.e[1][10]);
  Val x709 = (Val(1) - arg0.e[0][10]);
  Val x710 = (arg0.e[0][11] * arg0.e[1][11]);
  Val x711 = (Val(1) - arg0.e[0][11]);
  Val x712 = (arg0.e[0][12] * arg0.e[1][12]);
  Val x713 = (Val(1) - arg0.e[0][12]);
  Val x714 = (arg0.e[0][13] * arg0.e[1][13]);
  Val x715 = (Val(1) - arg0.e[0][13]);
  Val x716 = (arg0.e[0][14] * arg0.e[1][14]);
  Val x717 = (Val(1) - arg0.e[0][14]);
  Val x718 = (arg0.e[0][15] * arg0.e[1][15]);
  Val x719 = (Val(1) - arg0.e[0][15]);
  Val x720 = (arg0.e[0][16] * arg0.e[1][16]);
  Val x721 = (Val(1) - arg0.e[0][16]);
  Val x722 = (arg0.e[0][17] * arg0.e[1][17]);
  Val x723 = (Val(1) - arg0.e[0][17]);
  Val x724 = (arg0.e[0][18] * arg0.e[1][18]);
  Val x725 = (Val(1) - arg0.e[0][18]);
  Val x726 = (arg0.e[0][19] * arg0.e[1][19]);
  Val x727 = (Val(1) - arg0.e[0][19]);
  Val x728 = (arg0.e[0][20] * arg0.e[1][20]);
  Val x729 = (Val(1) - arg0.e[0][20]);
  Val x730 = (arg0.e[0][21] * arg0.e[1][21]);
  Val x731 = (Val(1) - arg0.e[0][21]);
  Val x732 = (arg0.e[0][22] * arg0.e[1][22]);
  Val x733 = (Val(1) - arg0.e[0][22]);
  Val x734 = (arg0.e[0][23] * arg0.e[1][23]);
  Val x735 = (Val(1) - arg0.e[0][23]);
  Val x736 = (arg0.e[0][24] * arg0.e[1][24]);
  Val x737 = (Val(1) - arg0.e[0][24]);
  Val x738 = (arg0.e[0][25] * arg0.e[1][25]);
  Val x739 = (Val(1) - arg0.e[0][25]);
  Val x740 = (arg0.e[0][26] * arg0.e[1][26]);
  Val x741 = (Val(1) - arg0.e[0][26]);
  Val x742 = (arg0.e[0][27] * arg0.e[1][27]);
  Val x743 = (Val(1) - arg0.e[0][27]);
  Val x744 = (arg0.e[0][28] * arg0.e[1][28]);
  Val x745 = (Val(1) - arg0.e[0][28]);
  Val x746 = (arg0.e[0][29] * arg0.e[1][29]);
  Val x747 = (Val(1) - arg0.e[0][29]);
  Val x748 = (arg0.e[0][30] * arg0.e[1][30]);
  Val x749 = (Val(1) - arg0.e[0][30]);
  Val x750 = (arg0.e[0][31] * arg0.e[1][31]);
  Val x751 = (Val(1) - arg0.e[0][31]);
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x752 = ((x688 + (x689 * arg0.e[2][0])) + ((x690 + (x691 * arg0.e[2][1])) * Val(2)));
  Val x753 = ((x752 + ((x692 + (x693 * arg0.e[2][2])) * Val(4))) +
              ((x694 + (x695 * arg0.e[2][3])) * Val(8)));
  Val x754 = ((x753 + ((x696 + (x697 * arg0.e[2][4])) * Val(16))) +
              ((x698 + (x699 * arg0.e[2][5])) * Val(32)));
  Val x755 = ((x754 + ((x700 + (x701 * arg0.e[2][6])) * Val(64))) +
              ((x702 + (x703 * arg0.e[2][7])) * Val(128)));
  Val x756 = ((x755 + ((x704 + (x705 * arg0.e[2][8])) * Val(256))) +
              ((x706 + (x707 * arg0.e[2][9])) * Val(512)));
  Val x757 = ((x756 + ((x708 + (x709 * arg0.e[2][10])) * Val(1024))) +
              ((x710 + (x711 * arg0.e[2][11])) * Val(2048)));
  Val x758 = ((x757 + ((x712 + (x713 * arg0.e[2][12])) * Val(4096))) +
              ((x714 + (x715 * arg0.e[2][13])) * Val(8192)));
  Val x759 = ((x758 + ((x716 + (x717 * arg0.e[2][14])) * Val(16384))) +
              ((x718 + (x719 * arg0.e[2][15])) * Val(32768)));
  Val x760 = ((x720 + (x721 * arg0.e[2][16])) + ((x722 + (x723 * arg0.e[2][17])) * Val(2)));
  Val x761 = ((x760 + ((x724 + (x725 * arg0.e[2][18])) * Val(4))) +
              ((x726 + (x727 * arg0.e[2][19])) * Val(8)));
  Val x762 = ((x761 + ((x728 + (x729 * arg0.e[2][20])) * Val(16))) +
              ((x730 + (x731 * arg0.e[2][21])) * Val(32)));
  Val x763 = ((x762 + ((x732 + (x733 * arg0.e[2][22])) * Val(64))) +
              ((x734 + (x735 * arg0.e[2][23])) * Val(128)));
  Val x764 = ((x763 + ((x736 + (x737 * arg0.e[2][24])) * Val(256))) +
              ((x738 + (x739 * arg0.e[2][25])) * Val(512)));
  Val x765 = ((x764 + ((x740 + (x741 * arg0.e[2][26])) * Val(1024))) +
              ((x742 + (x743 * arg0.e[2][27])) * Val(2048)));
  Val x766 = ((x765 + ((x744 + (x745 * arg0.e[2][28])) * Val(4096))) +
              ((x746 + (x747 * arg0.e[2][29])) * Val(8192)));
  Val x767 = ((x766 + ((x748 + (x749 * arg0.e[2][30])) * Val(16384))) +
              ((x750 + (x751 * arg0.e[2][31])) * Val(32768)));
  Val x768 = ((x576 - (x562 * x482)) + ((x577 - (x565 * x485)) * Val(2)));
  Val x769 = ((x768 + ((x578 - (x568 * x488)) * Val(4))) + ((x579 - (x571 * x491)) * Val(8)));
  Val x770 = ((x769 + ((x580 - (x574 * x494)) * Val(16))) + ((x581 - (x481 * x497)) * Val(32)));
  Val x771 = ((x770 + ((x582 - (x484 * x500)) * Val(64))) + ((x583 - (x487 * x503)) * Val(128)));
  Val x772 = ((x771 + ((x584 - (x490 * x506)) * Val(256))) + ((x585 - (x493 * x509)) * Val(512)));
  Val x773 = ((x772 + ((x586 - (x496 * x512)) * Val(1024))) + ((x587 - (x499 * x515)) * Val(2048)));
  Val x774 = ((x773 + ((x588 - (x502 * x518)) * Val(4096))) + ((x589 - (x505 * x521)) * Val(8192)));
  Val x775 =
      ((x774 + ((x590 - (x508 * x524)) * Val(16384))) + ((x591 - (x511 * x527)) * Val(32768)));
  Val x776 = ((x592 - (x514 * x530)) + ((x593 - (x517 * x533)) * Val(2)));
  Val x777 = ((x776 + ((x594 - (x520 * x536)) * Val(4))) + ((x595 - (x523 * x539)) * Val(8)));
  Val x778 = ((x777 + ((x596 - (x526 * x542)) * Val(16))) + ((x597 - (x529 * x545)) * Val(32)));
  Val x779 = ((x778 + ((x598 - (x532 * x548)) * Val(64))) + ((x599 - (x535 * x551)) * Val(128)));
  Val x780 = ((x779 + ((x600 - (x538 * x554)) * Val(256))) + ((x601 - (x541 * x557)) * Val(512)));
  Val x781 = ((x780 + ((x602 - (x544 * x560)) * Val(1024))) + ((x603 - (x547 * x563)) * Val(2048)));
  Val x782 = ((x781 + ((x604 - (x550 * x566)) * Val(4096))) + ((x605 - (x553 * x569)) * Val(8192)));
  Val x783 =
      ((x782 + ((x606 - (x556 * x572)) * Val(16384))) + ((x607 - (x559 * x575)) * Val(32768)));
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x784 = (((x667 + x661) + x662) + (x759 + x775));
  Val x785 = (((x687 + x681) + x682) + (x767 + x783));
  Val x786 = (((x627 + x621) + x622) + (arg1[0] + x784));
  Val x787 = (((x647 + x641) + x642) + (arg1[1] + x785));
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:75)
  Val x788 = (arg0.a[0][0] * arg0.a[1][0]);
  Val x789 = (Val(1) - arg0.a[2][0]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x790 = (Val(1) - arg0.a[1][0]);
  Val x791 = (arg0.a[0][0] * x790);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x792 = (Val(1) - arg0.a[0][0]);
  Val x793 = ((x792 * arg0.a[1][0]) * arg0.a[2][0]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x794 = (((x788 * x789) + (x791 * arg0.a[2][0])) + x793);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x795 = (arg0.a[0][1] * arg0.a[1][1]);
  Val x796 = (Val(1) - arg0.a[2][1]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x797 = (Val(1) - arg0.a[1][1]);
  Val x798 = (arg0.a[0][1] * x797);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x799 = (Val(1) - arg0.a[0][1]);
  Val x800 = ((x799 * arg0.a[1][1]) * arg0.a[2][1]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x801 = (((x795 * x796) + (x798 * arg0.a[2][1])) + x800);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x802 = (arg0.a[0][2] * arg0.a[1][2]);
  Val x803 = (Val(1) - arg0.a[2][2]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x804 = (Val(1) - arg0.a[1][2]);
  Val x805 = (arg0.a[0][2] * x804);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x806 = (Val(1) - arg0.a[0][2]);
  Val x807 = ((x806 * arg0.a[1][2]) * arg0.a[2][2]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x808 = (((x802 * x803) + (x805 * arg0.a[2][2])) + x807);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x809 = (arg0.a[0][3] * arg0.a[1][3]);
  Val x810 = (Val(1) - arg0.a[2][3]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x811 = (Val(1) - arg0.a[1][3]);
  Val x812 = (arg0.a[0][3] * x811);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x813 = (Val(1) - arg0.a[0][3]);
  Val x814 = ((x813 * arg0.a[1][3]) * arg0.a[2][3]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x815 = (((x809 * x810) + (x812 * arg0.a[2][3])) + x814);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x816 = (arg0.a[0][4] * arg0.a[1][4]);
  Val x817 = (Val(1) - arg0.a[2][4]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x818 = (Val(1) - arg0.a[1][4]);
  Val x819 = (arg0.a[0][4] * x818);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x820 = (Val(1) - arg0.a[0][4]);
  Val x821 = ((x820 * arg0.a[1][4]) * arg0.a[2][4]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x822 = (((x816 * x817) + (x819 * arg0.a[2][4])) + x821);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x823 = (arg0.a[0][5] * arg0.a[1][5]);
  Val x824 = (Val(1) - arg0.a[2][5]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x825 = (Val(1) - arg0.a[1][5]);
  Val x826 = (arg0.a[0][5] * x825);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x827 = (Val(1) - arg0.a[0][5]);
  Val x828 = ((x827 * arg0.a[1][5]) * arg0.a[2][5]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x829 = (((x823 * x824) + (x826 * arg0.a[2][5])) + x828);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x830 = (arg0.a[0][6] * arg0.a[1][6]);
  Val x831 = (Val(1) - arg0.a[2][6]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x832 = (Val(1) - arg0.a[1][6]);
  Val x833 = (arg0.a[0][6] * x832);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x834 = (Val(1) - arg0.a[0][6]);
  Val x835 = ((x834 * arg0.a[1][6]) * arg0.a[2][6]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x836 = (((x830 * x831) + (x833 * arg0.a[2][6])) + x835);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x837 = (arg0.a[0][7] * arg0.a[1][7]);
  Val x838 = (Val(1) - arg0.a[2][7]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x839 = (Val(1) - arg0.a[1][7]);
  Val x840 = (arg0.a[0][7] * x839);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x841 = (Val(1) - arg0.a[0][7]);
  Val x842 = ((x841 * arg0.a[1][7]) * arg0.a[2][7]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x843 = (((x837 * x838) + (x840 * arg0.a[2][7])) + x842);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x844 = (arg0.a[0][8] * arg0.a[1][8]);
  Val x845 = (Val(1) - arg0.a[2][8]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x846 = (Val(1) - arg0.a[1][8]);
  Val x847 = (arg0.a[0][8] * x846);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x848 = (Val(1) - arg0.a[0][8]);
  Val x849 = ((x848 * arg0.a[1][8]) * arg0.a[2][8]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x850 = (((x844 * x845) + (x847 * arg0.a[2][8])) + x849);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x851 = (arg0.a[0][9] * arg0.a[1][9]);
  Val x852 = (Val(1) - arg0.a[2][9]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x853 = (Val(1) - arg0.a[1][9]);
  Val x854 = (arg0.a[0][9] * x853);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x855 = (Val(1) - arg0.a[0][9]);
  Val x856 = ((x855 * arg0.a[1][9]) * arg0.a[2][9]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x857 = (((x851 * x852) + (x854 * arg0.a[2][9])) + x856);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x858 = (arg0.a[0][10] * arg0.a[1][10]);
  Val x859 = (Val(1) - arg0.a[2][10]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x860 = (Val(1) - arg0.a[1][10]);
  Val x861 = (arg0.a[0][10] * x860);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x862 = (Val(1) - arg0.a[0][10]);
  Val x863 = ((x862 * arg0.a[1][10]) * arg0.a[2][10]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x864 = (((x858 * x859) + (x861 * arg0.a[2][10])) + x863);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x865 = (arg0.a[0][11] * arg0.a[1][11]);
  Val x866 = (Val(1) - arg0.a[2][11]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x867 = (Val(1) - arg0.a[1][11]);
  Val x868 = (arg0.a[0][11] * x867);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x869 = (Val(1) - arg0.a[0][11]);
  Val x870 = ((x869 * arg0.a[1][11]) * arg0.a[2][11]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x871 = (((x865 * x866) + (x868 * arg0.a[2][11])) + x870);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x872 = (arg0.a[0][12] * arg0.a[1][12]);
  Val x873 = (Val(1) - arg0.a[2][12]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x874 = (Val(1) - arg0.a[1][12]);
  Val x875 = (arg0.a[0][12] * x874);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x876 = (Val(1) - arg0.a[0][12]);
  Val x877 = ((x876 * arg0.a[1][12]) * arg0.a[2][12]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x878 = (((x872 * x873) + (x875 * arg0.a[2][12])) + x877);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x879 = (arg0.a[0][13] * arg0.a[1][13]);
  Val x880 = (Val(1) - arg0.a[2][13]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x881 = (Val(1) - arg0.a[1][13]);
  Val x882 = (arg0.a[0][13] * x881);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x883 = (Val(1) - arg0.a[0][13]);
  Val x884 = ((x883 * arg0.a[1][13]) * arg0.a[2][13]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x885 = (((x879 * x880) + (x882 * arg0.a[2][13])) + x884);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x886 = (arg0.a[0][14] * arg0.a[1][14]);
  Val x887 = (Val(1) - arg0.a[2][14]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x888 = (Val(1) - arg0.a[1][14]);
  Val x889 = (arg0.a[0][14] * x888);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x890 = (Val(1) - arg0.a[0][14]);
  Val x891 = ((x890 * arg0.a[1][14]) * arg0.a[2][14]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x892 = (((x886 * x887) + (x889 * arg0.a[2][14])) + x891);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x893 = (arg0.a[0][15] * arg0.a[1][15]);
  Val x894 = (Val(1) - arg0.a[2][15]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x895 = (Val(1) - arg0.a[1][15]);
  Val x896 = (arg0.a[0][15] * x895);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x897 = (Val(1) - arg0.a[0][15]);
  Val x898 = ((x897 * arg0.a[1][15]) * arg0.a[2][15]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x899 = (((x893 * x894) + (x896 * arg0.a[2][15])) + x898);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x900 = (arg0.a[0][16] * arg0.a[1][16]);
  Val x901 = (Val(1) - arg0.a[2][16]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x902 = (Val(1) - arg0.a[1][16]);
  Val x903 = (arg0.a[0][16] * x902);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x904 = (Val(1) - arg0.a[0][16]);
  Val x905 = ((x904 * arg0.a[1][16]) * arg0.a[2][16]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x906 = (((x900 * x901) + (x903 * arg0.a[2][16])) + x905);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x907 = (arg0.a[0][17] * arg0.a[1][17]);
  Val x908 = (Val(1) - arg0.a[2][17]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x909 = (Val(1) - arg0.a[1][17]);
  Val x910 = (arg0.a[0][17] * x909);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x911 = (Val(1) - arg0.a[0][17]);
  Val x912 = ((x911 * arg0.a[1][17]) * arg0.a[2][17]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x913 = (((x907 * x908) + (x910 * arg0.a[2][17])) + x912);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x914 = (arg0.a[0][18] * arg0.a[1][18]);
  Val x915 = (Val(1) - arg0.a[2][18]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x916 = (Val(1) - arg0.a[1][18]);
  Val x917 = (arg0.a[0][18] * x916);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x918 = (Val(1) - arg0.a[0][18]);
  Val x919 = ((x918 * arg0.a[1][18]) * arg0.a[2][18]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x920 = (((x914 * x915) + (x917 * arg0.a[2][18])) + x919);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x921 = (arg0.a[0][19] * arg0.a[1][19]);
  Val x922 = (Val(1) - arg0.a[2][19]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x923 = (Val(1) - arg0.a[1][19]);
  Val x924 = (arg0.a[0][19] * x923);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x925 = (Val(1) - arg0.a[0][19]);
  Val x926 = ((x925 * arg0.a[1][19]) * arg0.a[2][19]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x927 = (((x921 * x922) + (x924 * arg0.a[2][19])) + x926);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x928 = (arg0.a[0][20] * arg0.a[1][20]);
  Val x929 = (Val(1) - arg0.a[2][20]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x930 = (Val(1) - arg0.a[1][20]);
  Val x931 = (arg0.a[0][20] * x930);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x932 = (Val(1) - arg0.a[0][20]);
  Val x933 = ((x932 * arg0.a[1][20]) * arg0.a[2][20]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x934 = (((x928 * x929) + (x931 * arg0.a[2][20])) + x933);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x935 = (arg0.a[0][21] * arg0.a[1][21]);
  Val x936 = (Val(1) - arg0.a[2][21]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x937 = (Val(1) - arg0.a[1][21]);
  Val x938 = (arg0.a[0][21] * x937);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x939 = (Val(1) - arg0.a[0][21]);
  Val x940 = ((x939 * arg0.a[1][21]) * arg0.a[2][21]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x941 = (((x935 * x936) + (x938 * arg0.a[2][21])) + x940);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x942 = (arg0.a[0][22] * arg0.a[1][22]);
  Val x943 = (Val(1) - arg0.a[2][22]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x944 = (Val(1) - arg0.a[1][22]);
  Val x945 = (arg0.a[0][22] * x944);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x946 = (Val(1) - arg0.a[0][22]);
  Val x947 = ((x946 * arg0.a[1][22]) * arg0.a[2][22]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x948 = (((x942 * x943) + (x945 * arg0.a[2][22])) + x947);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x949 = (arg0.a[0][23] * arg0.a[1][23]);
  Val x950 = (Val(1) - arg0.a[2][23]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x951 = (Val(1) - arg0.a[1][23]);
  Val x952 = (arg0.a[0][23] * x951);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x953 = (Val(1) - arg0.a[0][23]);
  Val x954 = ((x953 * arg0.a[1][23]) * arg0.a[2][23]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x955 = (((x949 * x950) + (x952 * arg0.a[2][23])) + x954);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x956 = (arg0.a[0][24] * arg0.a[1][24]);
  Val x957 = (Val(1) - arg0.a[2][24]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x958 = (Val(1) - arg0.a[1][24]);
  Val x959 = (arg0.a[0][24] * x958);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x960 = (Val(1) - arg0.a[0][24]);
  Val x961 = ((x960 * arg0.a[1][24]) * arg0.a[2][24]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x962 = (((x956 * x957) + (x959 * arg0.a[2][24])) + x961);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x963 = (arg0.a[0][25] * arg0.a[1][25]);
  Val x964 = (Val(1) - arg0.a[2][25]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x965 = (Val(1) - arg0.a[1][25]);
  Val x966 = (arg0.a[0][25] * x965);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x967 = (Val(1) - arg0.a[0][25]);
  Val x968 = ((x967 * arg0.a[1][25]) * arg0.a[2][25]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x969 = (((x963 * x964) + (x966 * arg0.a[2][25])) + x968);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x970 = (arg0.a[0][26] * arg0.a[1][26]);
  Val x971 = (Val(1) - arg0.a[2][26]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x972 = (Val(1) - arg0.a[1][26]);
  Val x973 = (arg0.a[0][26] * x972);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x974 = (Val(1) - arg0.a[0][26]);
  Val x975 = ((x974 * arg0.a[1][26]) * arg0.a[2][26]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x976 = (((x970 * x971) + (x973 * arg0.a[2][26])) + x975);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x977 = (arg0.a[0][27] * arg0.a[1][27]);
  Val x978 = (Val(1) - arg0.a[2][27]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x979 = (Val(1) - arg0.a[1][27]);
  Val x980 = (arg0.a[0][27] * x979);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x981 = (Val(1) - arg0.a[0][27]);
  Val x982 = ((x981 * arg0.a[1][27]) * arg0.a[2][27]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x983 = (((x977 * x978) + (x980 * arg0.a[2][27])) + x982);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x984 = (arg0.a[0][28] * arg0.a[1][28]);
  Val x985 = (Val(1) - arg0.a[2][28]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x986 = (Val(1) - arg0.a[1][28]);
  Val x987 = (arg0.a[0][28] * x986);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x988 = (Val(1) - arg0.a[0][28]);
  Val x989 = ((x988 * arg0.a[1][28]) * arg0.a[2][28]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x990 = (((x984 * x985) + (x987 * arg0.a[2][28])) + x989);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x991 = (arg0.a[0][29] * arg0.a[1][29]);
  Val x992 = (Val(1) - arg0.a[2][29]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x993 = (Val(1) - arg0.a[1][29]);
  Val x994 = (arg0.a[0][29] * x993);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x995 = (Val(1) - arg0.a[0][29]);
  Val x996 = ((x995 * arg0.a[1][29]) * arg0.a[2][29]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x997 = (((x991 * x992) + (x994 * arg0.a[2][29])) + x996);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x998 = (arg0.a[0][30] * arg0.a[1][30]);
  Val x999 = (Val(1) - arg0.a[2][30]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1000 = (Val(1) - arg0.a[1][30]);
  Val x1001 = (arg0.a[0][30] * x1000);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x1002 = (Val(1) - arg0.a[0][30]);
  Val x1003 = ((x1002 * arg0.a[1][30]) * arg0.a[2][30]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1004 = (((x998 * x999) + (x1001 * arg0.a[2][30])) + x1003);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x1005 = (arg0.a[0][31] * arg0.a[1][31]);
  Val x1006 = (Val(1) - arg0.a[2][31]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1007 = (Val(1) - arg0.a[1][31]);
  Val x1008 = (arg0.a[0][31] * x1007);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x1009 = (Val(1) - arg0.a[0][31]);
  Val x1010 = ((x1009 * arg0.a[1][31]) * arg0.a[2][31]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1011 = (((x1005 * x1006) + (x1008 * arg0.a[2][31])) + x1010);
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x1012 = ((x794 + (x788 * arg0.a[2][0])) + ((x801 + (x795 * arg0.a[2][1])) * Val(2)));
  Val x1013 = ((x1012 + ((x808 + (x802 * arg0.a[2][2])) * Val(4))) +
               ((x815 + (x809 * arg0.a[2][3])) * Val(8)));
  Val x1014 = ((x1013 + ((x822 + (x816 * arg0.a[2][4])) * Val(16))) +
               ((x829 + (x823 * arg0.a[2][5])) * Val(32)));
  Val x1015 = ((x1014 + ((x836 + (x830 * arg0.a[2][6])) * Val(64))) +
               ((x843 + (x837 * arg0.a[2][7])) * Val(128)));
  Val x1016 = ((x1015 + ((x850 + (x844 * arg0.a[2][8])) * Val(256))) +
               ((x857 + (x851 * arg0.a[2][9])) * Val(512)));
  Val x1017 = ((x1016 + ((x864 + (x858 * arg0.a[2][10])) * Val(1024))) +
               ((x871 + (x865 * arg0.a[2][11])) * Val(2048)));
  Val x1018 = ((x1017 + ((x878 + (x872 * arg0.a[2][12])) * Val(4096))) +
               ((x885 + (x879 * arg0.a[2][13])) * Val(8192)));
  Val x1019 = ((x1018 + ((x892 + (x886 * arg0.a[2][14])) * Val(16384))) +
               ((x899 + (x893 * arg0.a[2][15])) * Val(32768)));
  Val x1020 = ((x906 + (x900 * arg0.a[2][16])) + ((x913 + (x907 * arg0.a[2][17])) * Val(2)));
  Val x1021 = ((x1020 + ((x920 + (x914 * arg0.a[2][18])) * Val(4))) +
               ((x927 + (x921 * arg0.a[2][19])) * Val(8)));
  Val x1022 = ((x1021 + ((x934 + (x928 * arg0.a[2][20])) * Val(16))) +
               ((x941 + (x935 * arg0.a[2][21])) * Val(32)));
  Val x1023 = ((x1022 + ((x948 + (x942 * arg0.a[2][22])) * Val(64))) +
               ((x955 + (x949 * arg0.a[2][23])) * Val(128)));
  Val x1024 = ((x1023 + ((x962 + (x956 * arg0.a[2][24])) * Val(256))) +
               ((x969 + (x963 * arg0.a[2][25])) * Val(512)));
  Val x1025 = ((x1024 + ((x976 + (x970 * arg0.a[2][26])) * Val(1024))) +
               ((x983 + (x977 * arg0.a[2][27])) * Val(2048)));
  Val x1026 = ((x1025 + ((x990 + (x984 * arg0.a[2][28])) * Val(4096))) +
               ((x997 + (x991 * arg0.a[2][29])) * Val(8192)));
  Val x1027 = ((x1026 + ((x1004 + (x998 * arg0.a[2][30])) * Val(16384))) +
               ((x1011 + (x1005 * arg0.a[2][31])) * Val(32768)));
  Val x1028 = ((x448 - (x416 * x354)) + ((x449 - (x419 * x357)) * Val(2)));
  Val x1029 = ((x1028 + ((x450 - (x422 * x360)) * Val(4))) + ((x451 - (x425 * x363)) * Val(8)));
  Val x1030 = ((x1029 + ((x452 - (x428 * x366)) * Val(16))) + ((x453 - (x431 * x369)) * Val(32)));
  Val x1031 = ((x1030 + ((x454 - (x434 * x372)) * Val(64))) + ((x455 - (x437 * x375)) * Val(128)));
  Val x1032 = ((x1031 + ((x456 - (x440 * x378)) * Val(256))) + ((x457 - (x443 * x381)) * Val(512)));
  Val x1033 =
      ((x1032 + ((x458 - (x446 * x384)) * Val(1024))) + ((x459 - (x353 * x387)) * Val(2048)));
  Val x1034 =
      ((x1033 + ((x460 - (x356 * x390)) * Val(4096))) + ((x461 - (x359 * x393)) * Val(8192)));
  Val x1035 =
      ((x1034 + ((x462 - (x362 * x396)) * Val(16384))) + ((x463 - (x365 * x399)) * Val(32768)));
  Val x1036 = ((x464 - (x368 * x402)) + ((x465 - (x371 * x405)) * Val(2)));
  Val x1037 = ((x1036 + ((x466 - (x374 * x408)) * Val(4))) + ((x467 - (x377 * x411)) * Val(8)));
  Val x1038 = ((x1037 + ((x468 - (x380 * x414)) * Val(16))) + ((x469 - (x383 * x417)) * Val(32)));
  Val x1039 = ((x1038 + ((x470 - (x386 * x420)) * Val(64))) + ((x471 - (x389 * x423)) * Val(128)));
  Val x1040 = ((x1039 + ((x472 - (x392 * x426)) * Val(256))) + ((x473 - (x395 * x429)) * Val(512)));
  Val x1041 =
      ((x1040 + ((x474 - (x398 * x432)) * Val(1024))) + ((x475 - (x401 * x435)) * Val(2048)));
  Val x1042 =
      ((x1041 + ((x476 - (x404 * x438)) * Val(4096))) + ((x477 - (x407 * x441)) * Val(8192)));
  Val x1043 =
      ((x1042 + ((x478 - (x410 * x444)) * Val(16384))) + ((x479 - (x413 * x447)) * Val(32768)));
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:76)
  Val x1044 = (arg0.a[3][1] * Val(2));
  Val x1045 = (arg0.a[3][2] * Val(4));
  Val x1046 = (arg0.a[3][3] * Val(8));
  Val x1047 = (arg0.a[3][4] * Val(16));
  Val x1048 = (arg0.a[3][5] * Val(32));
  Val x1049 = (arg0.a[3][6] * Val(64));
  Val x1050 = (arg0.a[3][7] * Val(128));
  Val x1051 = (arg0.a[3][8] * Val(256));
  Val x1052 = (arg0.a[3][9] * Val(512));
  Val x1053 = (arg0.a[3][10] * Val(1024));
  Val x1054 = (arg0.a[3][11] * Val(2048));
  Val x1055 = (arg0.a[3][12] * Val(4096));
  Val x1056 = (arg0.a[3][13] * Val(8192));
  Val x1057 = (arg0.a[3][14] * Val(16384));
  Val x1058 = (arg0.a[3][15] * Val(32768));
  Val x1059 = (arg0.a[3][0] + x1044);
  Val x1060 = (((x1059 + x1045) + x1046) + x1047);
  Val x1061 = (((x1060 + x1048) + x1049) + x1050);
  Val x1062 = (((x1061 + x1051) + x1052) + x1053);
  Val x1063 = (((x1062 + x1054) + x1055) + x1056);
  Val x1064 = (arg0.a[3][17] * Val(2));
  Val x1065 = (arg0.a[3][18] * Val(4));
  Val x1066 = (arg0.a[3][19] * Val(8));
  Val x1067 = (arg0.a[3][20] * Val(16));
  Val x1068 = (arg0.a[3][21] * Val(32));
  Val x1069 = (arg0.a[3][22] * Val(64));
  Val x1070 = (arg0.a[3][23] * Val(128));
  Val x1071 = (arg0.a[3][24] * Val(256));
  Val x1072 = (arg0.a[3][25] * Val(512));
  Val x1073 = (arg0.a[3][26] * Val(1024));
  Val x1074 = (arg0.a[3][27] * Val(2048));
  Val x1075 = (arg0.a[3][28] * Val(4096));
  Val x1076 = (arg0.a[3][29] * Val(8192));
  Val x1077 = (arg0.a[3][30] * Val(16384));
  Val x1078 = (arg0.a[3][31] * Val(32768));
  Val x1079 = (arg0.a[3][16] + x1064);
  Val x1080 = (((x1079 + x1065) + x1066) + x1067);
  Val x1081 = (((x1080 + x1068) + x1069) + x1070);
  Val x1082 = (((x1081 + x1071) + x1072) + x1073);
  Val x1083 = (((x1082 + x1074) + x1075) + x1076);
  // DoShaStep(zirgen/circuit/keccak2/top.zir:142)
  UnpackReg_32__16_Struct x1084 =
      exec_CarryAndExpand(ctx,
                          Val2Array{(x786 + (x1019 + x1035)), (x787 + (x1027 + x1043))},
                          LAYOUT_LOOKUP(layout4, a));
  // DoShaStep(zirgen/circuit/keccak2/top.zir:143)
  UnpackReg_32__16_Struct x1085 = exec_CarryAndExpand(
      ctx,
      Val2Array{(x786 + ((x1063 + x1057) + x1058)), (x787 + ((x1083 + x1077) + x1078))},
      LAYOUT_LOOKUP(layout4, e));
  // DoShaStep(zirgen/circuit/keccak2/top.zir:142)
  Val32Array x1086 = Val32Array{x1084._super[0]._super._super,  x1084._super[1]._super._super,
                                x1084._super[2]._super._super,  x1084._super[3]._super._super,
                                x1084._super[4]._super._super,  x1084._super[5]._super._super,
                                x1084._super[6]._super._super,  x1084._super[7]._super._super,
                                x1084._super[8]._super._super,  x1084._super[9]._super._super,
                                x1084._super[10]._super._super, x1084._super[11]._super._super,
                                x1084._super[12]._super._super, x1084._super[13]._super._super,
                                x1084._super[14]._super._super, x1084._super[15]._super._super,
                                x1084._super[16]._super._super, x1084._super[17]._super._super,
                                x1084._super[18]._super._super, x1084._super[19]._super._super,
                                x1084._super[20]._super._super, x1084._super[21]._super._super,
                                x1084._super[22]._super._super, x1084._super[23]._super._super,
                                x1084._super[24]._super._super, x1084._super[25]._super._super,
                                x1084._super[26]._super._super, x1084._super[27]._super._super,
                                x1084._super[28]._super._super, x1084._super[29]._super._super,
                                x1084._super[30]._super._super, x1084._super[31]._super._super};
  // DoShaStep(zirgen/circuit/keccak2/top.zir:143)
  Val32Array x1087 = Val32Array{x1085._super[0]._super._super,  x1085._super[1]._super._super,
                                x1085._super[2]._super._super,  x1085._super[3]._super._super,
                                x1085._super[4]._super._super,  x1085._super[5]._super._super,
                                x1085._super[6]._super._super,  x1085._super[7]._super._super,
                                x1085._super[8]._super._super,  x1085._super[9]._super._super,
                                x1085._super[10]._super._super, x1085._super[11]._super._super,
                                x1085._super[12]._super._super, x1085._super[13]._super._super,
                                x1085._super[14]._super._super, x1085._super[15]._super._super,
                                x1085._super[16]._super._super, x1085._super[17]._super._super,
                                x1085._super[18]._super._super, x1085._super[19]._super._super,
                                x1085._super[20]._super._super, x1085._super[21]._super._super,
                                x1085._super[22]._super._super, x1085._super[23]._super._super,
                                x1085._super[24]._super._super, x1085._super[25]._super._super,
                                x1085._super[26]._super._super, x1085._super[27]._super._super,
                                x1085._super[28]._super._super, x1085._super[29]._super._super,
                                x1085._super[30]._super._super, x1085._super[31]._super._super};
  // DoShaStep(zirgen/circuit/keccak2/top.zir:140)
  Val32Array x1088 = Val32Array{
      x351._super[0]._super._super,  x351._super[1]._super._super,  x351._super[2]._super._super,
      x351._super[3]._super._super,  x351._super[4]._super._super,  x351._super[5]._super._super,
      x351._super[6]._super._super,  x351._super[7]._super._super,  x351._super[8]._super._super,
      x351._super[9]._super._super,  x351._super[10]._super._super, x351._super[11]._super._super,
      x351._super[12]._super._super, x351._super[13]._super._super, x351._super[14]._super._super,
      x351._super[15]._super._super, x351._super[16]._super._super, x351._super[17]._super._super,
      x351._super[18]._super._super, x351._super[19]._super._super, x351._super[20]._super._super,
      x351._super[21]._super._super, x351._super[22]._super._super, x351._super[23]._super._super,
      x351._super[24]._super._super, x351._super[25]._super._super, x351._super[26]._super._super,
      x351._super[27]._super._super, x351._super[28]._super._super, x351._super[29]._super._super,
      x351._super[30]._super._super, x351._super[31]._super._super};
  return DoShaStepStruct{
      .w = x351,
      .a = x1084,
      .e = x1085,
      .newState = ShaStateStruct{.a = Val32Array4Array{x1086, arg0.a[0], arg0.a[1], arg0.a[2]},
                                 .e = Val32Array4Array{x1087, arg0.e[0], arg0.e[1], arg0.e[2]},
                                 .w = Val32Array16Array{x1088,
                                                        arg0.w[0],
                                                        arg0.w[1],
                                                        arg0.w[2],
                                                        arg0.w[3],
                                                        arg0.w[4],
                                                        arg0.w[5],
                                                        arg0.w[6],
                                                        arg0.w[7],
                                                        arg0.w[8],
                                                        arg0.w[9],
                                                        arg0.w[10],
                                                        arg0.w[11],
                                                        arg0.w[12],
                                                        arg0.w[13],
                                                        arg0.w[14]}}};
}
__device__ ControlStateStruct exec_ShaNextRound(ExecContext& ctx,
                                                ControlStateStruct arg0,
                                                BoundLayout<ShaNextRoundLayout> layout1) {
  // ShaNextRound(zirgen/circuit/keccak2/top.zir:419)
  Val x2 = (arg0.round._super - Val(7));
  NondetRegStruct x3 = exec_IsZero(ctx, x2, LAYOUT_LOOKUP(layout1, isLast));
  // ShaNextRound(zirgen/circuit/keccak2/top.zir:423)
  Val x4 = (arg0.round._super + Val(1));
  ControlStateStruct x5;
  if (to_size_t(x3._super)) {
    // ShaNextRound(zirgen/circuit/keccak2/top.zir:421)
    ControlStateStruct x6 = exec_ControlState(ctx,
                                              Val(10),
                                              arg0.subType._super,
                                              arg0.block._super,
                                              Val(0),
                                              LAYOUT_LOOKUP(layout1, _super.arm0));
    x5 = x6;
  } else if (to_size_t((Val(1) - x3._super))) {
    // ShaNextRound(zirgen/circuit/keccak2/top.zir:423)
    ControlStateStruct x7 = exec_ControlState(ctx,
                                              Val(9),
                                              arg0.subType._super,
                                              arg0.block._super,
                                              x4,
                                              LAYOUT_LOOKUP(layout1, _super.arm1));
    x5 = x7;
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  // ShaNextRound(zirgen/circuit/keccak2/top.zir:420)
  ControlStateStruct x8 = back_ControlState(ctx, 0, LAYOUT_LOOKUP(layout1, _super._super));
  return x8;
}

} // namespace risc0::circuit::keccak::cuda
