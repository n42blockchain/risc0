#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"

namespace risc0::impl {

__device__ TopStateStruct exec_ShutdownCycle(ExecContext& ctx,
                                             TopStateStruct arg0,
                                             BoundLayout<TopStateLayout> layout1,
                                             GlobalBuf global2) {
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:394)
  BoundLayout<_globalLayout> x3 = BIND_LAYOUT(kLayoutGlobal, global2);
  // Log(<preamble>:22)
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:395)
  INVOKE_EXTERN(ctx, log, "ShutdownCycle", std::initializer_list<Val>{});
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:394)
  NondetRegStruct16Array x4 = arg0.sflat;
  Val16Array x5 = Val16Array{x4[0]._super,
                             x4[1]._super,
                             x4[2]._super,
                             x4[3]._super,
                             x4[4]._super,
                             x4[5]._super,
                             x4[6]._super,
                             x4[7]._super,
                             x4[8]._super,
                             x4[9]._super,
                             x4[10]._super,
                             x4[11]._super,
                             x4[12]._super,
                             x4[13]._super,
                             x4[14]._super,
                             x4[15]._super};
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:396)
  DigestRegStruct x6 = exec_DigestReg(ctx, x5, LAYOUT_LOOKUP(x3, finalDigest));
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:399)
  TopStateStruct x7 = exec_TopState(
      ctx,
      Val800Array{
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val100Array{Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      x5,
      layout1);
  return x7;
}

} // namespace risc0::impl
