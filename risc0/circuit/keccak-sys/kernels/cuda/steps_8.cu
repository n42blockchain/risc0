#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"

namespace risc0::impl {

__device__ TopStateStruct exec_ExpandCycle(ExecContext& ctx,
                                           TopStateStruct arg0,
                                           Val arg1,
                                           BoundLayout<ExpandCycleLayout> layout2) {
  // Log(<preamble>:22)
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:347)
  INVOKE_EXTERN(ctx, log, "ExpandCycle", std::initializer_list<Val>{});
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:346)
  TopStateKflatStruct100Array x3 = arg0.kflat;
  TopStateKflatStruct50Array x4;
  if (to_size_t(arg1)) {
    x4 = TopStateKflatStruct50Array{
        x3[2],  x3[3],  x3[6],  x3[7],  x3[10], x3[11], x3[14], x3[15], x3[18], x3[19],
        x3[22], x3[23], x3[26], x3[27], x3[30], x3[31], x3[34], x3[35], x3[38], x3[39],
        x3[42], x3[43], x3[46], x3[47], x3[50], x3[51], x3[54], x3[55], x3[58], x3[59],
        x3[62], x3[63], x3[66], x3[67], x3[70], x3[71], x3[74], x3[75], x3[78], x3[79],
        x3[82], x3[83], x3[86], x3[87], x3[90], x3[91], x3[94], x3[95], x3[98], x3[99]};
  } else if (to_size_t((Val(1) - arg1))) {
    x4 = TopStateKflatStruct50Array{
        x3[0],  x3[1],  x3[4],  x3[5],  x3[8],  x3[9],  x3[12], x3[13], x3[16], x3[17],
        x3[20], x3[21], x3[24], x3[25], x3[28], x3[29], x3[32], x3[33], x3[36], x3[37],
        x3[40], x3[41], x3[44], x3[45], x3[48], x3[49], x3[52], x3[53], x3[56], x3[57],
        x3[60], x3[61], x3[64], x3[65], x3[68], x3[69], x3[72], x3[73], x3[76], x3[77],
        x3[80], x3[81], x3[84], x3[85], x3[88], x3[89], x3[92], x3[93], x3[96], x3[97]};
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:349)
  Val x5 = x4[0]._super._super;
  Val x6 = x4[1]._super._super;
  Val x7 = x4[2]._super._super;
  Val x8 = x4[3]._super._super;
  Val x9 = x4[4]._super._super;
  Val x10 = x4[5]._super._super;
  Val x11 = x4[6]._super._super;
  Val x12 = x4[7]._super._super;
  Val x13 = x4[8]._super._super;
  Val x14 = x4[9]._super._super;
  Val x15 = x4[10]._super._super;
  Val x16 = x4[11]._super._super;
  Val x17 = x4[12]._super._super;
  Val x18 = x4[13]._super._super;
  Val x19 = x4[14]._super._super;
  Val x20 = x4[15]._super._super;
  Val x21 = x4[16]._super._super;
  Val x22 = x4[17]._super._super;
  Val x23 = x4[18]._super._super;
  Val x24 = x4[19]._super._super;
  Val x25 = x4[20]._super._super;
  Val x26 = x4[21]._super._super;
  Val x27 = x4[22]._super._super;
  Val x28 = x4[23]._super._super;
  Val x29 = x4[24]._super._super;
  Val x30 = x4[25]._super._super;
  Val x31 = x4[26]._super._super;
  Val x32 = x4[27]._super._super;
  Val x33 = x4[28]._super._super;
  Val x34 = x4[29]._super._super;
  Val x35 = x4[30]._super._super;
  Val x36 = x4[31]._super._super;
  Val x37 = x4[32]._super._super;
  Val x38 = x4[33]._super._super;
  Val x39 = x4[34]._super._super;
  Val x40 = x4[35]._super._super;
  Val x41 = x4[36]._super._super;
  Val x42 = x4[37]._super._super;
  Val x43 = x4[38]._super._super;
  Val x44 = x4[39]._super._super;
  Val x45 = x4[40]._super._super;
  Val x46 = x4[41]._super._super;
  Val x47 = x4[42]._super._super;
  Val x48 = x4[43]._super._super;
  Val x49 = x4[44]._super._super;
  Val x50 = x4[45]._super._super;
  Val x51 = x4[46]._super._super;
  Val x52 = x4[47]._super._super;
  Val x53 = x4[48]._super._super;
  Val x54 = x4[49]._super._super;
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:348)
  UnpackReg_800__16_Struct x55 = exec_UnpackReg_800__16_(
      ctx,
      Val50Array{x5,  x6,  x7,  x8,  x9,  x10, x11, x12, x13, x14, x15, x16, x17,
                 x18, x19, x20, x21, x22, x23, x24, x25, x26, x27, x28, x29, x30,
                 x31, x32, x33, x34, x35, x36, x37, x38, x39, x40, x41, x42, x43,
                 x44, x45, x46, x47, x48, x49, x50, x51, x52, x53, x54},
      LAYOUT_LOOKUP(layout2, bits));
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:346)
  NondetRegStruct16Array x56 = arg0.sflat;
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:348)
  UnpackReg_800__16__SuperStruct800Array x57 = x55._super;
  Val x58 = x57[0]._super._super;
  Val x59 = x57[1]._super._super;
  Val x60 = x57[2]._super._super;
  Val x61 = x57[3]._super._super;
  Val x62 = x57[4]._super._super;
  Val x63 = x57[5]._super._super;
  Val x64 = x57[6]._super._super;
  Val x65 = x57[7]._super._super;
  Val x66 = x57[8]._super._super;
  Val x67 = x57[9]._super._super;
  Val x68 = x57[10]._super._super;
  Val x69 = x57[11]._super._super;
  Val x70 = x57[12]._super._super;
  Val x71 = x57[13]._super._super;
  Val x72 = x57[14]._super._super;
  Val x73 = x57[15]._super._super;
  Val x74 = x57[16]._super._super;
  Val x75 = x57[17]._super._super;
  Val x76 = x57[18]._super._super;
  Val x77 = x57[19]._super._super;
  Val x78 = x57[20]._super._super;
  Val x79 = x57[21]._super._super;
  Val x80 = x57[22]._super._super;
  Val x81 = x57[23]._super._super;
  Val x82 = x57[24]._super._super;
  Val x83 = x57[25]._super._super;
  Val x84 = x57[26]._super._super;
  Val x85 = x57[27]._super._super;
  Val x86 = x57[28]._super._super;
  Val x87 = x57[29]._super._super;
  Val x88 = x57[30]._super._super;
  Val x89 = x57[31]._super._super;
  Val x90 = x57[32]._super._super;
  Val x91 = x57[33]._super._super;
  Val x92 = x57[34]._super._super;
  Val x93 = x57[35]._super._super;
  Val x94 = x57[36]._super._super;
  Val x95 = x57[37]._super._super;
  Val x96 = x57[38]._super._super;
  Val x97 = x57[39]._super._super;
  Val x98 = x57[40]._super._super;
  Val x99 = x57[41]._super._super;
  Val x100 = x57[42]._super._super;
  Val x101 = x57[43]._super._super;
  Val x102 = x57[44]._super._super;
  Val x103 = x57[45]._super._super;
  Val x104 = x57[46]._super._super;
  Val x105 = x57[47]._super._super;
  Val x106 = x57[48]._super._super;
  Val x107 = x57[49]._super._super;
  Val x108 = x57[50]._super._super;
  Val x109 = x57[51]._super._super;
  Val x110 = x57[52]._super._super;
  Val x111 = x57[53]._super._super;
  Val x112 = x57[54]._super._super;
  Val x113 = x57[55]._super._super;
  Val x114 = x57[56]._super._super;
  Val x115 = x57[57]._super._super;
  Val x116 = x57[58]._super._super;
  Val x117 = x57[59]._super._super;
  Val x118 = x57[60]._super._super;
  Val x119 = x57[61]._super._super;
  Val x120 = x57[62]._super._super;
  Val x121 = x57[63]._super._super;
  Val x122 = x57[64]._super._super;
  Val x123 = x57[65]._super._super;
  Val x124 = x57[66]._super._super;
  Val x125 = x57[67]._super._super;
  Val x126 = x57[68]._super._super;
  Val x127 = x57[69]._super._super;
  Val x128 = x57[70]._super._super;
  Val x129 = x57[71]._super._super;
  Val x130 = x57[72]._super._super;
  Val x131 = x57[73]._super._super;
  Val x132 = x57[74]._super._super;
  Val x133 = x57[75]._super._super;
  Val x134 = x57[76]._super._super;
  Val x135 = x57[77]._super._super;
  Val x136 = x57[78]._super._super;
  Val x137 = x57[79]._super._super;
  Val x138 = x57[80]._super._super;
  Val x139 = x57[81]._super._super;
  Val x140 = x57[82]._super._super;
  Val x141 = x57[83]._super._super;
  Val x142 = x57[84]._super._super;
  Val x143 = x57[85]._super._super;
  Val x144 = x57[86]._super._super;
  Val x145 = x57[87]._super._super;
  Val x146 = x57[88]._super._super;
  Val x147 = x57[89]._super._super;
  Val x148 = x57[90]._super._super;
  Val x149 = x57[91]._super._super;
  Val x150 = x57[92]._super._super;
  Val x151 = x57[93]._super._super;
  Val x152 = x57[94]._super._super;
  Val x153 = x57[95]._super._super;
  Val x154 = x57[96]._super._super;
  Val x155 = x57[97]._super._super;
  Val x156 = x57[98]._super._super;
  Val x157 = x57[99]._super._super;
  Val x158 = x57[100]._super._super;
  Val x159 = x57[101]._super._super;
  Val x160 = x57[102]._super._super;
  Val x161 = x57[103]._super._super;
  Val x162 = x57[104]._super._super;
  Val x163 = x57[105]._super._super;
  Val x164 = x57[106]._super._super;
  Val x165 = x57[107]._super._super;
  Val x166 = x57[108]._super._super;
  Val x167 = x57[109]._super._super;
  Val x168 = x57[110]._super._super;
  Val x169 = x57[111]._super._super;
  Val x170 = x57[112]._super._super;
  Val x171 = x57[113]._super._super;
  Val x172 = x57[114]._super._super;
  Val x173 = x57[115]._super._super;
  Val x174 = x57[116]._super._super;
  Val x175 = x57[117]._super._super;
  Val x176 = x57[118]._super._super;
  Val x177 = x57[119]._super._super;
  Val x178 = x57[120]._super._super;
  Val x179 = x57[121]._super._super;
  Val x180 = x57[122]._super._super;
  Val x181 = x57[123]._super._super;
  Val x182 = x57[124]._super._super;
  Val x183 = x57[125]._super._super;
  Val x184 = x57[126]._super._super;
  Val x185 = x57[127]._super._super;
  Val x186 = x57[128]._super._super;
  Val x187 = x57[129]._super._super;
  Val x188 = x57[130]._super._super;
  Val x189 = x57[131]._super._super;
  Val x190 = x57[132]._super._super;
  Val x191 = x57[133]._super._super;
  Val x192 = x57[134]._super._super;
  Val x193 = x57[135]._super._super;
  Val x194 = x57[136]._super._super;
  Val x195 = x57[137]._super._super;
  Val x196 = x57[138]._super._super;
  Val x197 = x57[139]._super._super;
  Val x198 = x57[140]._super._super;
  Val x199 = x57[141]._super._super;
  Val x200 = x57[142]._super._super;
  Val x201 = x57[143]._super._super;
  Val x202 = x57[144]._super._super;
  Val x203 = x57[145]._super._super;
  Val x204 = x57[146]._super._super;
  Val x205 = x57[147]._super._super;
  Val x206 = x57[148]._super._super;
  Val x207 = x57[149]._super._super;
  Val x208 = x57[150]._super._super;
  Val x209 = x57[151]._super._super;
  Val x210 = x57[152]._super._super;
  Val x211 = x57[153]._super._super;
  Val x212 = x57[154]._super._super;
  Val x213 = x57[155]._super._super;
  Val x214 = x57[156]._super._super;
  Val x215 = x57[157]._super._super;
  Val x216 = x57[158]._super._super;
  Val x217 = x57[159]._super._super;
  Val x218 = x57[160]._super._super;
  Val x219 = x57[161]._super._super;
  Val x220 = x57[162]._super._super;
  Val x221 = x57[163]._super._super;
  Val x222 = x57[164]._super._super;
  Val x223 = x57[165]._super._super;
  Val x224 = x57[166]._super._super;
  Val x225 = x57[167]._super._super;
  Val x226 = x57[168]._super._super;
  Val x227 = x57[169]._super._super;
  Val x228 = x57[170]._super._super;
  Val x229 = x57[171]._super._super;
  Val x230 = x57[172]._super._super;
  Val x231 = x57[173]._super._super;
  Val x232 = x57[174]._super._super;
  Val x233 = x57[175]._super._super;
  Val x234 = x57[176]._super._super;
  Val x235 = x57[177]._super._super;
  Val x236 = x57[178]._super._super;
  Val x237 = x57[179]._super._super;
  Val x238 = x57[180]._super._super;
  Val x239 = x57[181]._super._super;
  Val x240 = x57[182]._super._super;
  Val x241 = x57[183]._super._super;
  Val x242 = x57[184]._super._super;
  Val x243 = x57[185]._super._super;
  Val x244 = x57[186]._super._super;
  Val x245 = x57[187]._super._super;
  Val x246 = x57[188]._super._super;
  Val x247 = x57[189]._super._super;
  Val x248 = x57[190]._super._super;
  Val x249 = x57[191]._super._super;
  Val x250 = x57[192]._super._super;
  Val x251 = x57[193]._super._super;
  Val x252 = x57[194]._super._super;
  Val x253 = x57[195]._super._super;
  Val x254 = x57[196]._super._super;
  Val x255 = x57[197]._super._super;
  Val x256 = x57[198]._super._super;
  Val x257 = x57[199]._super._super;
  Val x258 = x57[200]._super._super;
  Val x259 = x57[201]._super._super;
  Val x260 = x57[202]._super._super;
  Val x261 = x57[203]._super._super;
  Val x262 = x57[204]._super._super;
  Val x263 = x57[205]._super._super;
  Val x264 = x57[206]._super._super;
  Val x265 = x57[207]._super._super;
  Val x266 = x57[208]._super._super;
  Val x267 = x57[209]._super._super;
  Val x268 = x57[210]._super._super;
  Val x269 = x57[211]._super._super;
  Val x270 = x57[212]._super._super;
  Val x271 = x57[213]._super._super;
  Val x272 = x57[214]._super._super;
  Val x273 = x57[215]._super._super;
  Val x274 = x57[216]._super._super;
  Val x275 = x57[217]._super._super;
  Val x276 = x57[218]._super._super;
  Val x277 = x57[219]._super._super;
  Val x278 = x57[220]._super._super;
  Val x279 = x57[221]._super._super;
  Val x280 = x57[222]._super._super;
  Val x281 = x57[223]._super._super;
  Val x282 = x57[224]._super._super;
  Val x283 = x57[225]._super._super;
  Val x284 = x57[226]._super._super;
  Val x285 = x57[227]._super._super;
  Val x286 = x57[228]._super._super;
  Val x287 = x57[229]._super._super;
  Val x288 = x57[230]._super._super;
  Val x289 = x57[231]._super._super;
  Val x290 = x57[232]._super._super;
  Val x291 = x57[233]._super._super;
  Val x292 = x57[234]._super._super;
  Val x293 = x57[235]._super._super;
  Val x294 = x57[236]._super._super;
  Val x295 = x57[237]._super._super;
  Val x296 = x57[238]._super._super;
  Val x297 = x57[239]._super._super;
  Val x298 = x57[240]._super._super;
  Val x299 = x57[241]._super._super;
  Val x300 = x57[242]._super._super;
  Val x301 = x57[243]._super._super;
  Val x302 = x57[244]._super._super;
  Val x303 = x57[245]._super._super;
  Val x304 = x57[246]._super._super;
  Val x305 = x57[247]._super._super;
  Val x306 = x57[248]._super._super;
  Val x307 = x57[249]._super._super;
  Val x308 = x57[250]._super._super;
  Val x309 = x57[251]._super._super;
  Val x310 = x57[252]._super._super;
  Val x311 = x57[253]._super._super;
  Val x312 = x57[254]._super._super;
  Val x313 = x57[255]._super._super;
  Val x314 = x57[256]._super._super;
  Val x315 = x57[257]._super._super;
  Val x316 = x57[258]._super._super;
  Val x317 = x57[259]._super._super;
  Val x318 = x57[260]._super._super;
  Val x319 = x57[261]._super._super;
  Val x320 = x57[262]._super._super;
  Val x321 = x57[263]._super._super;
  Val x322 = x57[264]._super._super;
  Val x323 = x57[265]._super._super;
  Val x324 = x57[266]._super._super;
  Val x325 = x57[267]._super._super;
  Val x326 = x57[268]._super._super;
  Val x327 = x57[269]._super._super;
  Val x328 = x57[270]._super._super;
  Val x329 = x57[271]._super._super;
  Val x330 = x57[272]._super._super;
  Val x331 = x57[273]._super._super;
  Val x332 = x57[274]._super._super;
  Val x333 = x57[275]._super._super;
  Val x334 = x57[276]._super._super;
  Val x335 = x57[277]._super._super;
  Val x336 = x57[278]._super._super;
  Val x337 = x57[279]._super._super;
  Val x338 = x57[280]._super._super;
  Val x339 = x57[281]._super._super;
  Val x340 = x57[282]._super._super;
  Val x341 = x57[283]._super._super;
  Val x342 = x57[284]._super._super;
  Val x343 = x57[285]._super._super;
  Val x344 = x57[286]._super._super;
  Val x345 = x57[287]._super._super;
  Val x346 = x57[288]._super._super;
  Val x347 = x57[289]._super._super;
  Val x348 = x57[290]._super._super;
  Val x349 = x57[291]._super._super;
  Val x350 = x57[292]._super._super;
  Val x351 = x57[293]._super._super;
  Val x352 = x57[294]._super._super;
  Val x353 = x57[295]._super._super;
  Val x354 = x57[296]._super._super;
  Val x355 = x57[297]._super._super;
  Val x356 = x57[298]._super._super;
  Val x357 = x57[299]._super._super;
  Val x358 = x57[300]._super._super;
  Val x359 = x57[301]._super._super;
  Val x360 = x57[302]._super._super;
  Val x361 = x57[303]._super._super;
  Val x362 = x57[304]._super._super;
  Val x363 = x57[305]._super._super;
  Val x364 = x57[306]._super._super;
  Val x365 = x57[307]._super._super;
  Val x366 = x57[308]._super._super;
  Val x367 = x57[309]._super._super;
  Val x368 = x57[310]._super._super;
  Val x369 = x57[311]._super._super;
  Val x370 = x57[312]._super._super;
  Val x371 = x57[313]._super._super;
  Val x372 = x57[314]._super._super;
  Val x373 = x57[315]._super._super;
  Val x374 = x57[316]._super._super;
  Val x375 = x57[317]._super._super;
  Val x376 = x57[318]._super._super;
  Val x377 = x57[319]._super._super;
  Val x378 = x57[320]._super._super;
  Val x379 = x57[321]._super._super;
  Val x380 = x57[322]._super._super;
  Val x381 = x57[323]._super._super;
  Val x382 = x57[324]._super._super;
  Val x383 = x57[325]._super._super;
  Val x384 = x57[326]._super._super;
  Val x385 = x57[327]._super._super;
  Val x386 = x57[328]._super._super;
  Val x387 = x57[329]._super._super;
  Val x388 = x57[330]._super._super;
  Val x389 = x57[331]._super._super;
  Val x390 = x57[332]._super._super;
  Val x391 = x57[333]._super._super;
  Val x392 = x57[334]._super._super;
  Val x393 = x57[335]._super._super;
  Val x394 = x57[336]._super._super;
  Val x395 = x57[337]._super._super;
  Val x396 = x57[338]._super._super;
  Val x397 = x57[339]._super._super;
  Val x398 = x57[340]._super._super;
  Val x399 = x57[341]._super._super;
  Val x400 = x57[342]._super._super;
  Val x401 = x57[343]._super._super;
  Val x402 = x57[344]._super._super;
  Val x403 = x57[345]._super._super;
  Val x404 = x57[346]._super._super;
  Val x405 = x57[347]._super._super;
  Val x406 = x57[348]._super._super;
  Val x407 = x57[349]._super._super;
  Val x408 = x57[350]._super._super;
  Val x409 = x57[351]._super._super;
  Val x410 = x57[352]._super._super;
  Val x411 = x57[353]._super._super;
  Val x412 = x57[354]._super._super;
  Val x413 = x57[355]._super._super;
  Val x414 = x57[356]._super._super;
  Val x415 = x57[357]._super._super;
  Val x416 = x57[358]._super._super;
  Val x417 = x57[359]._super._super;
  Val x418 = x57[360]._super._super;
  Val x419 = x57[361]._super._super;
  Val x420 = x57[362]._super._super;
  Val x421 = x57[363]._super._super;
  Val x422 = x57[364]._super._super;
  Val x423 = x57[365]._super._super;
  Val x424 = x57[366]._super._super;
  Val x425 = x57[367]._super._super;
  Val x426 = x57[368]._super._super;
  Val x427 = x57[369]._super._super;
  Val x428 = x57[370]._super._super;
  Val x429 = x57[371]._super._super;
  Val x430 = x57[372]._super._super;
  Val x431 = x57[373]._super._super;
  Val x432 = x57[374]._super._super;
  Val x433 = x57[375]._super._super;
  Val x434 = x57[376]._super._super;
  Val x435 = x57[377]._super._super;
  Val x436 = x57[378]._super._super;
  Val x437 = x57[379]._super._super;
  Val x438 = x57[380]._super._super;
  Val x439 = x57[381]._super._super;
  Val x440 = x57[382]._super._super;
  Val x441 = x57[383]._super._super;
  Val x442 = x57[384]._super._super;
  Val x443 = x57[385]._super._super;
  Val x444 = x57[386]._super._super;
  Val x445 = x57[387]._super._super;
  Val x446 = x57[388]._super._super;
  Val x447 = x57[389]._super._super;
  Val x448 = x57[390]._super._super;
  Val x449 = x57[391]._super._super;
  Val x450 = x57[392]._super._super;
  Val x451 = x57[393]._super._super;
  Val x452 = x57[394]._super._super;
  Val x453 = x57[395]._super._super;
  Val x454 = x57[396]._super._super;
  Val x455 = x57[397]._super._super;
  Val x456 = x57[398]._super._super;
  Val x457 = x57[399]._super._super;
  Val x458 = x57[400]._super._super;
  Val x459 = x57[401]._super._super;
  Val x460 = x57[402]._super._super;
  Val x461 = x57[403]._super._super;
  Val x462 = x57[404]._super._super;
  Val x463 = x57[405]._super._super;
  Val x464 = x57[406]._super._super;
  Val x465 = x57[407]._super._super;
  Val x466 = x57[408]._super._super;
  Val x467 = x57[409]._super._super;
  Val x468 = x57[410]._super._super;
  Val x469 = x57[411]._super._super;
  Val x470 = x57[412]._super._super;
  Val x471 = x57[413]._super._super;
  Val x472 = x57[414]._super._super;
  Val x473 = x57[415]._super._super;
  Val x474 = x57[416]._super._super;
  Val x475 = x57[417]._super._super;
  Val x476 = x57[418]._super._super;
  Val x477 = x57[419]._super._super;
  Val x478 = x57[420]._super._super;
  Val x479 = x57[421]._super._super;
  Val x480 = x57[422]._super._super;
  Val x481 = x57[423]._super._super;
  Val x482 = x57[424]._super._super;
  Val x483 = x57[425]._super._super;
  Val x484 = x57[426]._super._super;
  Val x485 = x57[427]._super._super;
  Val x486 = x57[428]._super._super;
  Val x487 = x57[429]._super._super;
  Val x488 = x57[430]._super._super;
  Val x489 = x57[431]._super._super;
  Val x490 = x57[432]._super._super;
  Val x491 = x57[433]._super._super;
  Val x492 = x57[434]._super._super;
  Val x493 = x57[435]._super._super;
  Val x494 = x57[436]._super._super;
  Val x495 = x57[437]._super._super;
  Val x496 = x57[438]._super._super;
  Val x497 = x57[439]._super._super;
  Val x498 = x57[440]._super._super;
  Val x499 = x57[441]._super._super;
  Val x500 = x57[442]._super._super;
  Val x501 = x57[443]._super._super;
  Val x502 = x57[444]._super._super;
  Val x503 = x57[445]._super._super;
  Val x504 = x57[446]._super._super;
  Val x505 = x57[447]._super._super;
  Val x506 = x57[448]._super._super;
  Val x507 = x57[449]._super._super;
  Val x508 = x57[450]._super._super;
  Val x509 = x57[451]._super._super;
  Val x510 = x57[452]._super._super;
  Val x511 = x57[453]._super._super;
  Val x512 = x57[454]._super._super;
  Val x513 = x57[455]._super._super;
  Val x514 = x57[456]._super._super;
  Val x515 = x57[457]._super._super;
  Val x516 = x57[458]._super._super;
  Val x517 = x57[459]._super._super;
  Val x518 = x57[460]._super._super;
  Val x519 = x57[461]._super._super;
  Val x520 = x57[462]._super._super;
  Val x521 = x57[463]._super._super;
  Val x522 = x57[464]._super._super;
  Val x523 = x57[465]._super._super;
  Val x524 = x57[466]._super._super;
  Val x525 = x57[467]._super._super;
  Val x526 = x57[468]._super._super;
  Val x527 = x57[469]._super._super;
  Val x528 = x57[470]._super._super;
  Val x529 = x57[471]._super._super;
  Val x530 = x57[472]._super._super;
  Val x531 = x57[473]._super._super;
  Val x532 = x57[474]._super._super;
  Val x533 = x57[475]._super._super;
  Val x534 = x57[476]._super._super;
  Val x535 = x57[477]._super._super;
  Val x536 = x57[478]._super._super;
  Val x537 = x57[479]._super._super;
  Val x538 = x57[480]._super._super;
  Val x539 = x57[481]._super._super;
  Val x540 = x57[482]._super._super;
  Val x541 = x57[483]._super._super;
  Val x542 = x57[484]._super._super;
  Val x543 = x57[485]._super._super;
  Val x544 = x57[486]._super._super;
  Val x545 = x57[487]._super._super;
  Val x546 = x57[488]._super._super;
  Val x547 = x57[489]._super._super;
  Val x548 = x57[490]._super._super;
  Val x549 = x57[491]._super._super;
  Val x550 = x57[492]._super._super;
  Val x551 = x57[493]._super._super;
  Val x552 = x57[494]._super._super;
  Val x553 = x57[495]._super._super;
  Val x554 = x57[496]._super._super;
  Val x555 = x57[497]._super._super;
  Val x556 = x57[498]._super._super;
  Val x557 = x57[499]._super._super;
  Val x558 = x57[500]._super._super;
  Val x559 = x57[501]._super._super;
  Val x560 = x57[502]._super._super;
  Val x561 = x57[503]._super._super;
  Val x562 = x57[504]._super._super;
  Val x563 = x57[505]._super._super;
  Val x564 = x57[506]._super._super;
  Val x565 = x57[507]._super._super;
  Val x566 = x57[508]._super._super;
  Val x567 = x57[509]._super._super;
  Val x568 = x57[510]._super._super;
  Val x569 = x57[511]._super._super;
  Val x570 = x57[512]._super._super;
  Val x571 = x57[513]._super._super;
  Val x572 = x57[514]._super._super;
  Val x573 = x57[515]._super._super;
  Val x574 = x57[516]._super._super;
  Val x575 = x57[517]._super._super;
  Val x576 = x57[518]._super._super;
  Val x577 = x57[519]._super._super;
  Val x578 = x57[520]._super._super;
  Val x579 = x57[521]._super._super;
  Val x580 = x57[522]._super._super;
  Val x581 = x57[523]._super._super;
  Val x582 = x57[524]._super._super;
  Val x583 = x57[525]._super._super;
  Val x584 = x57[526]._super._super;
  Val x585 = x57[527]._super._super;
  Val x586 = x57[528]._super._super;
  Val x587 = x57[529]._super._super;
  Val x588 = x57[530]._super._super;
  Val x589 = x57[531]._super._super;
  Val x590 = x57[532]._super._super;
  Val x591 = x57[533]._super._super;
  Val x592 = x57[534]._super._super;
  Val x593 = x57[535]._super._super;
  Val x594 = x57[536]._super._super;
  Val x595 = x57[537]._super._super;
  Val x596 = x57[538]._super._super;
  Val x597 = x57[539]._super._super;
  Val x598 = x57[540]._super._super;
  Val x599 = x57[541]._super._super;
  Val x600 = x57[542]._super._super;
  Val x601 = x57[543]._super._super;
  Val x602 = x57[544]._super._super;
  Val x603 = x57[545]._super._super;
  Val x604 = x57[546]._super._super;
  Val x605 = x57[547]._super._super;
  Val x606 = x57[548]._super._super;
  Val x607 = x57[549]._super._super;
  Val x608 = x57[550]._super._super;
  Val x609 = x57[551]._super._super;
  Val x610 = x57[552]._super._super;
  Val x611 = x57[553]._super._super;
  Val x612 = x57[554]._super._super;
  Val x613 = x57[555]._super._super;
  Val x614 = x57[556]._super._super;
  Val x615 = x57[557]._super._super;
  Val x616 = x57[558]._super._super;
  Val x617 = x57[559]._super._super;
  Val x618 = x57[560]._super._super;
  Val x619 = x57[561]._super._super;
  Val x620 = x57[562]._super._super;
  Val x621 = x57[563]._super._super;
  Val x622 = x57[564]._super._super;
  Val x623 = x57[565]._super._super;
  Val x624 = x57[566]._super._super;
  Val x625 = x57[567]._super._super;
  Val x626 = x57[568]._super._super;
  Val x627 = x57[569]._super._super;
  Val x628 = x57[570]._super._super;
  Val x629 = x57[571]._super._super;
  Val x630 = x57[572]._super._super;
  Val x631 = x57[573]._super._super;
  Val x632 = x57[574]._super._super;
  Val x633 = x57[575]._super._super;
  Val x634 = x57[576]._super._super;
  Val x635 = x57[577]._super._super;
  Val x636 = x57[578]._super._super;
  Val x637 = x57[579]._super._super;
  Val x638 = x57[580]._super._super;
  Val x639 = x57[581]._super._super;
  Val x640 = x57[582]._super._super;
  Val x641 = x57[583]._super._super;
  Val x642 = x57[584]._super._super;
  Val x643 = x57[585]._super._super;
  Val x644 = x57[586]._super._super;
  Val x645 = x57[587]._super._super;
  Val x646 = x57[588]._super._super;
  Val x647 = x57[589]._super._super;
  Val x648 = x57[590]._super._super;
  Val x649 = x57[591]._super._super;
  Val x650 = x57[592]._super._super;
  Val x651 = x57[593]._super._super;
  Val x652 = x57[594]._super._super;
  Val x653 = x57[595]._super._super;
  Val x654 = x57[596]._super._super;
  Val x655 = x57[597]._super._super;
  Val x656 = x57[598]._super._super;
  Val x657 = x57[599]._super._super;
  Val x658 = x57[600]._super._super;
  Val x659 = x57[601]._super._super;
  Val x660 = x57[602]._super._super;
  Val x661 = x57[603]._super._super;
  Val x662 = x57[604]._super._super;
  Val x663 = x57[605]._super._super;
  Val x664 = x57[606]._super._super;
  Val x665 = x57[607]._super._super;
  Val x666 = x57[608]._super._super;
  Val x667 = x57[609]._super._super;
  Val x668 = x57[610]._super._super;
  Val x669 = x57[611]._super._super;
  Val x670 = x57[612]._super._super;
  Val x671 = x57[613]._super._super;
  Val x672 = x57[614]._super._super;
  Val x673 = x57[615]._super._super;
  Val x674 = x57[616]._super._super;
  Val x675 = x57[617]._super._super;
  Val x676 = x57[618]._super._super;
  Val x677 = x57[619]._super._super;
  Val x678 = x57[620]._super._super;
  Val x679 = x57[621]._super._super;
  Val x680 = x57[622]._super._super;
  Val x681 = x57[623]._super._super;
  Val x682 = x57[624]._super._super;
  Val x683 = x57[625]._super._super;
  Val x684 = x57[626]._super._super;
  Val x685 = x57[627]._super._super;
  Val x686 = x57[628]._super._super;
  Val x687 = x57[629]._super._super;
  Val x688 = x57[630]._super._super;
  Val x689 = x57[631]._super._super;
  Val x690 = x57[632]._super._super;
  Val x691 = x57[633]._super._super;
  Val x692 = x57[634]._super._super;
  Val x693 = x57[635]._super._super;
  Val x694 = x57[636]._super._super;
  Val x695 = x57[637]._super._super;
  Val x696 = x57[638]._super._super;
  Val x697 = x57[639]._super._super;
  Val x698 = x57[640]._super._super;
  Val x699 = x57[641]._super._super;
  Val x700 = x57[642]._super._super;
  Val x701 = x57[643]._super._super;
  Val x702 = x57[644]._super._super;
  Val x703 = x57[645]._super._super;
  Val x704 = x57[646]._super._super;
  Val x705 = x57[647]._super._super;
  Val x706 = x57[648]._super._super;
  Val x707 = x57[649]._super._super;
  Val x708 = x57[650]._super._super;
  Val x709 = x57[651]._super._super;
  Val x710 = x57[652]._super._super;
  Val x711 = x57[653]._super._super;
  Val x712 = x57[654]._super._super;
  Val x713 = x57[655]._super._super;
  Val x714 = x57[656]._super._super;
  Val x715 = x57[657]._super._super;
  Val x716 = x57[658]._super._super;
  Val x717 = x57[659]._super._super;
  Val x718 = x57[660]._super._super;
  Val x719 = x57[661]._super._super;
  Val x720 = x57[662]._super._super;
  Val x721 = x57[663]._super._super;
  Val x722 = x57[664]._super._super;
  Val x723 = x57[665]._super._super;
  Val x724 = x57[666]._super._super;
  Val x725 = x57[667]._super._super;
  Val x726 = x57[668]._super._super;
  Val x727 = x57[669]._super._super;
  Val x728 = x57[670]._super._super;
  Val x729 = x57[671]._super._super;
  Val x730 = x57[672]._super._super;
  Val x731 = x57[673]._super._super;
  Val x732 = x57[674]._super._super;
  Val x733 = x57[675]._super._super;
  Val x734 = x57[676]._super._super;
  Val x735 = x57[677]._super._super;
  Val x736 = x57[678]._super._super;
  Val x737 = x57[679]._super._super;
  Val x738 = x57[680]._super._super;
  Val x739 = x57[681]._super._super;
  Val x740 = x57[682]._super._super;
  Val x741 = x57[683]._super._super;
  Val x742 = x57[684]._super._super;
  Val x743 = x57[685]._super._super;
  Val x744 = x57[686]._super._super;
  Val x745 = x57[687]._super._super;
  Val x746 = x57[688]._super._super;
  Val x747 = x57[689]._super._super;
  Val x748 = x57[690]._super._super;
  Val x749 = x57[691]._super._super;
  Val x750 = x57[692]._super._super;
  Val x751 = x57[693]._super._super;
  Val x752 = x57[694]._super._super;
  Val x753 = x57[695]._super._super;
  Val x754 = x57[696]._super._super;
  Val x755 = x57[697]._super._super;
  Val x756 = x57[698]._super._super;
  Val x757 = x57[699]._super._super;
  Val x758 = x57[700]._super._super;
  Val x759 = x57[701]._super._super;
  Val x760 = x57[702]._super._super;
  Val x761 = x57[703]._super._super;
  Val x762 = x57[704]._super._super;
  Val x763 = x57[705]._super._super;
  Val x764 = x57[706]._super._super;
  Val x765 = x57[707]._super._super;
  Val x766 = x57[708]._super._super;
  Val x767 = x57[709]._super._super;
  Val x768 = x57[710]._super._super;
  Val x769 = x57[711]._super._super;
  Val x770 = x57[712]._super._super;
  Val x771 = x57[713]._super._super;
  Val x772 = x57[714]._super._super;
  Val x773 = x57[715]._super._super;
  Val x774 = x57[716]._super._super;
  Val x775 = x57[717]._super._super;
  Val x776 = x57[718]._super._super;
  Val x777 = x57[719]._super._super;
  Val x778 = x57[720]._super._super;
  Val x779 = x57[721]._super._super;
  Val x780 = x57[722]._super._super;
  Val x781 = x57[723]._super._super;
  Val x782 = x57[724]._super._super;
  Val x783 = x57[725]._super._super;
  Val x784 = x57[726]._super._super;
  Val x785 = x57[727]._super._super;
  Val x786 = x57[728]._super._super;
  Val x787 = x57[729]._super._super;
  Val x788 = x57[730]._super._super;
  Val x789 = x57[731]._super._super;
  Val x790 = x57[732]._super._super;
  Val x791 = x57[733]._super._super;
  Val x792 = x57[734]._super._super;
  Val x793 = x57[735]._super._super;
  Val x794 = x57[736]._super._super;
  Val x795 = x57[737]._super._super;
  Val x796 = x57[738]._super._super;
  Val x797 = x57[739]._super._super;
  Val x798 = x57[740]._super._super;
  Val x799 = x57[741]._super._super;
  Val x800 = x57[742]._super._super;
  Val x801 = x57[743]._super._super;
  Val x802 = x57[744]._super._super;
  Val x803 = x57[745]._super._super;
  Val x804 = x57[746]._super._super;
  Val x805 = x57[747]._super._super;
  Val x806 = x57[748]._super._super;
  Val x807 = x57[749]._super._super;
  Val x808 = x57[750]._super._super;
  Val x809 = x57[751]._super._super;
  Val x810 = x57[752]._super._super;
  Val x811 = x57[753]._super._super;
  Val x812 = x57[754]._super._super;
  Val x813 = x57[755]._super._super;
  Val x814 = x57[756]._super._super;
  Val x815 = x57[757]._super._super;
  Val x816 = x57[758]._super._super;
  Val x817 = x57[759]._super._super;
  Val x818 = x57[760]._super._super;
  Val x819 = x57[761]._super._super;
  Val x820 = x57[762]._super._super;
  Val x821 = x57[763]._super._super;
  Val x822 = x57[764]._super._super;
  Val x823 = x57[765]._super._super;
  Val x824 = x57[766]._super._super;
  Val x825 = x57[767]._super._super;
  Val x826 = x57[768]._super._super;
  Val x827 = x57[769]._super._super;
  Val x828 = x57[770]._super._super;
  Val x829 = x57[771]._super._super;
  Val x830 = x57[772]._super._super;
  Val x831 = x57[773]._super._super;
  Val x832 = x57[774]._super._super;
  Val x833 = x57[775]._super._super;
  Val x834 = x57[776]._super._super;
  Val x835 = x57[777]._super._super;
  Val x836 = x57[778]._super._super;
  Val x837 = x57[779]._super._super;
  Val x838 = x57[780]._super._super;
  Val x839 = x57[781]._super._super;
  Val x840 = x57[782]._super._super;
  Val x841 = x57[783]._super._super;
  Val x842 = x57[784]._super._super;
  Val x843 = x57[785]._super._super;
  Val x844 = x57[786]._super._super;
  Val x845 = x57[787]._super._super;
  Val x846 = x57[788]._super._super;
  Val x847 = x57[789]._super._super;
  Val x848 = x57[790]._super._super;
  Val x849 = x57[791]._super._super;
  Val x850 = x57[792]._super._super;
  Val x851 = x57[793]._super._super;
  Val x852 = x57[794]._super._super;
  Val x853 = x57[795]._super._super;
  Val x854 = x57[796]._super._super;
  Val x855 = x57[797]._super._super;
  Val x856 = x57[798]._super._super;
  Val x857 = x57[799]._super._super;
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:346)
  Val x858 = x3[0]._super._super;
  Val x859 = x3[1]._super._super;
  Val x860 = x3[2]._super._super;
  Val x861 = x3[3]._super._super;
  Val x862 = x3[4]._super._super;
  Val x863 = x3[5]._super._super;
  Val x864 = x3[6]._super._super;
  Val x865 = x3[7]._super._super;
  Val x866 = x3[8]._super._super;
  Val x867 = x3[9]._super._super;
  Val x868 = x3[10]._super._super;
  Val x869 = x3[11]._super._super;
  Val x870 = x3[12]._super._super;
  Val x871 = x3[13]._super._super;
  Val x872 = x3[14]._super._super;
  Val x873 = x3[15]._super._super;
  Val x874 = x3[16]._super._super;
  Val x875 = x3[17]._super._super;
  Val x876 = x3[18]._super._super;
  Val x877 = x3[19]._super._super;
  Val x878 = x3[20]._super._super;
  Val x879 = x3[21]._super._super;
  Val x880 = x3[22]._super._super;
  Val x881 = x3[23]._super._super;
  Val x882 = x3[24]._super._super;
  Val x883 = x3[25]._super._super;
  Val x884 = x3[26]._super._super;
  Val x885 = x3[27]._super._super;
  Val x886 = x3[28]._super._super;
  Val x887 = x3[29]._super._super;
  Val x888 = x3[30]._super._super;
  Val x889 = x3[31]._super._super;
  Val x890 = x3[32]._super._super;
  Val x891 = x3[33]._super._super;
  Val x892 = x3[34]._super._super;
  Val x893 = x3[35]._super._super;
  Val x894 = x3[36]._super._super;
  Val x895 = x3[37]._super._super;
  Val x896 = x3[38]._super._super;
  Val x897 = x3[39]._super._super;
  Val x898 = x3[40]._super._super;
  Val x899 = x3[41]._super._super;
  Val x900 = x3[42]._super._super;
  Val x901 = x3[43]._super._super;
  Val x902 = x3[44]._super._super;
  Val x903 = x3[45]._super._super;
  Val x904 = x3[46]._super._super;
  Val x905 = x3[47]._super._super;
  Val x906 = x3[48]._super._super;
  Val x907 = x3[49]._super._super;
  Val x908 = x3[50]._super._super;
  Val x909 = x3[51]._super._super;
  Val x910 = x3[52]._super._super;
  Val x911 = x3[53]._super._super;
  Val x912 = x3[54]._super._super;
  Val x913 = x3[55]._super._super;
  Val x914 = x3[56]._super._super;
  Val x915 = x3[57]._super._super;
  Val x916 = x3[58]._super._super;
  Val x917 = x3[59]._super._super;
  Val x918 = x3[60]._super._super;
  Val x919 = x3[61]._super._super;
  Val x920 = x3[62]._super._super;
  Val x921 = x3[63]._super._super;
  Val x922 = x3[64]._super._super;
  Val x923 = x3[65]._super._super;
  Val x924 = x3[66]._super._super;
  Val x925 = x3[67]._super._super;
  Val x926 = x3[68]._super._super;
  Val x927 = x3[69]._super._super;
  Val x928 = x3[70]._super._super;
  Val x929 = x3[71]._super._super;
  Val x930 = x3[72]._super._super;
  Val x931 = x3[73]._super._super;
  Val x932 = x3[74]._super._super;
  Val x933 = x3[75]._super._super;
  Val x934 = x3[76]._super._super;
  Val x935 = x3[77]._super._super;
  Val x936 = x3[78]._super._super;
  Val x937 = x3[79]._super._super;
  Val x938 = x3[80]._super._super;
  Val x939 = x3[81]._super._super;
  Val x940 = x3[82]._super._super;
  Val x941 = x3[83]._super._super;
  Val x942 = x3[84]._super._super;
  Val x943 = x3[85]._super._super;
  Val x944 = x3[86]._super._super;
  Val x945 = x3[87]._super._super;
  Val x946 = x3[88]._super._super;
  Val x947 = x3[89]._super._super;
  Val x948 = x3[90]._super._super;
  Val x949 = x3[91]._super._super;
  Val x950 = x3[92]._super._super;
  Val x951 = x3[93]._super._super;
  Val x952 = x3[94]._super._super;
  Val x953 = x3[95]._super._super;
  Val x954 = x3[96]._super._super;
  Val x955 = x3[97]._super._super;
  Val x956 = x3[98]._super._super;
  Val x957 = x3[99]._super._super;
  Val16Array x958 = Val16Array{x56[0]._super,
                               x56[1]._super,
                               x56[2]._super,
                               x56[3]._super,
                               x56[4]._super,
                               x56[5]._super,
                               x56[6]._super,
                               x56[7]._super,
                               x56[8]._super,
                               x56[9]._super,
                               x56[10]._super,
                               x56[11]._super,
                               x56[12]._super,
                               x56[13]._super,
                               x56[14]._super,
                               x56[15]._super};
  // ExpandCycle(zirgen/circuit/keccak2/top.zir:363)
  TopStateStruct x959 = exec_TopState(
      ctx,
      Val800Array{
          x58,  x59,  x60,  x61,  x62,  x63,  x64,  x65,  x66,  x67,  x68,  x69,  x70,  x71,  x72,
          x73,  x74,  x75,  x76,  x77,  x78,  x79,  x80,  x81,  x82,  x83,  x84,  x85,  x86,  x87,
          x88,  x89,  x90,  x91,  x92,  x93,  x94,  x95,  x96,  x97,  x98,  x99,  x100, x101, x102,
          x103, x104, x105, x106, x107, x108, x109, x110, x111, x112, x113, x114, x115, x116, x117,
          x118, x119, x120, x121, x122, x123, x124, x125, x126, x127, x128, x129, x130, x131, x132,
          x133, x134, x135, x136, x137, x138, x139, x140, x141, x142, x143, x144, x145, x146, x147,
          x148, x149, x150, x151, x152, x153, x154, x155, x156, x157, x158, x159, x160, x161, x162,
          x163, x164, x165, x166, x167, x168, x169, x170, x171, x172, x173, x174, x175, x176, x177,
          x178, x179, x180, x181, x182, x183, x184, x185, x186, x187, x188, x189, x190, x191, x192,
          x193, x194, x195, x196, x197, x198, x199, x200, x201, x202, x203, x204, x205, x206, x207,
          x208, x209, x210, x211, x212, x213, x214, x215, x216, x217, x218, x219, x220, x221, x222,
          x223, x224, x225, x226, x227, x228, x229, x230, x231, x232, x233, x234, x235, x236, x237,
          x238, x239, x240, x241, x242, x243, x244, x245, x246, x247, x248, x249, x250, x251, x252,
          x253, x254, x255, x256, x257, x258, x259, x260, x261, x262, x263, x264, x265, x266, x267,
          x268, x269, x270, x271, x272, x273, x274, x275, x276, x277, x278, x279, x280, x281, x282,
          x283, x284, x285, x286, x287, x288, x289, x290, x291, x292, x293, x294, x295, x296, x297,
          x298, x299, x300, x301, x302, x303, x304, x305, x306, x307, x308, x309, x310, x311, x312,
          x313, x314, x315, x316, x317, x318, x319, x320, x321, x322, x323, x324, x325, x326, x327,
          x328, x329, x330, x331, x332, x333, x334, x335, x336, x337, x338, x339, x340, x341, x342,
          x343, x344, x345, x346, x347, x348, x349, x350, x351, x352, x353, x354, x355, x356, x357,
          x358, x359, x360, x361, x362, x363, x364, x365, x366, x367, x368, x369, x370, x371, x372,
          x373, x374, x375, x376, x377, x378, x379, x380, x381, x382, x383, x384, x385, x386, x387,
          x388, x389, x390, x391, x392, x393, x394, x395, x396, x397, x398, x399, x400, x401, x402,
          x403, x404, x405, x406, x407, x408, x409, x410, x411, x412, x413, x414, x415, x416, x417,
          x418, x419, x420, x421, x422, x423, x424, x425, x426, x427, x428, x429, x430, x431, x432,
          x433, x434, x435, x436, x437, x438, x439, x440, x441, x442, x443, x444, x445, x446, x447,
          x448, x449, x450, x451, x452, x453, x454, x455, x456, x457, x458, x459, x460, x461, x462,
          x463, x464, x465, x466, x467, x468, x469, x470, x471, x472, x473, x474, x475, x476, x477,
          x478, x479, x480, x481, x482, x483, x484, x485, x486, x487, x488, x489, x490, x491, x492,
          x493, x494, x495, x496, x497, x498, x499, x500, x501, x502, x503, x504, x505, x506, x507,
          x508, x509, x510, x511, x512, x513, x514, x515, x516, x517, x518, x519, x520, x521, x522,
          x523, x524, x525, x526, x527, x528, x529, x530, x531, x532, x533, x534, x535, x536, x537,
          x538, x539, x540, x541, x542, x543, x544, x545, x546, x547, x548, x549, x550, x551, x552,
          x553, x554, x555, x556, x557, x558, x559, x560, x561, x562, x563, x564, x565, x566, x567,
          x568, x569, x570, x571, x572, x573, x574, x575, x576, x577, x578, x579, x580, x581, x582,
          x583, x584, x585, x586, x587, x588, x589, x590, x591, x592, x593, x594, x595, x596, x597,
          x598, x599, x600, x601, x602, x603, x604, x605, x606, x607, x608, x609, x610, x611, x612,
          x613, x614, x615, x616, x617, x618, x619, x620, x621, x622, x623, x624, x625, x626, x627,
          x628, x629, x630, x631, x632, x633, x634, x635, x636, x637, x638, x639, x640, x641, x642,
          x643, x644, x645, x646, x647, x648, x649, x650, x651, x652, x653, x654, x655, x656, x657,
          x658, x659, x660, x661, x662, x663, x664, x665, x666, x667, x668, x669, x670, x671, x672,
          x673, x674, x675, x676, x677, x678, x679, x680, x681, x682, x683, x684, x685, x686, x687,
          x688, x689, x690, x691, x692, x693, x694, x695, x696, x697, x698, x699, x700, x701, x702,
          x703, x704, x705, x706, x707, x708, x709, x710, x711, x712, x713, x714, x715, x716, x717,
          x718, x719, x720, x721, x722, x723, x724, x725, x726, x727, x728, x729, x730, x731, x732,
          x733, x734, x735, x736, x737, x738, x739, x740, x741, x742, x743, x744, x745, x746, x747,
          x748, x749, x750, x751, x752, x753, x754, x755, x756, x757, x758, x759, x760, x761, x762,
          x763, x764, x765, x766, x767, x768, x769, x770, x771, x772, x773, x774, x775, x776, x777,
          x778, x779, x780, x781, x782, x783, x784, x785, x786, x787, x788, x789, x790, x791, x792,
          x793, x794, x795, x796, x797, x798, x799, x800, x801, x802, x803, x804, x805, x806, x807,
          x808, x809, x810, x811, x812, x813, x814, x815, x816, x817, x818, x819, x820, x821, x822,
          x823, x824, x825, x826, x827, x828, x829, x830, x831, x832, x833, x834, x835, x836, x837,
          x838, x839, x840, x841, x842, x843, x844, x845, x846, x847, x848, x849, x850, x851, x852,
          x853, x854, x855, x856, x857},
      Val100Array{x858, x859, x860, x861, x862, x863, x864, x865, x866, x867, x868, x869, x870,
                  x871, x872, x873, x874, x875, x876, x877, x878, x879, x880, x881, x882, x883,
                  x884, x885, x886, x887, x888, x889, x890, x891, x892, x893, x894, x895, x896,
                  x897, x898, x899, x900, x901, x902, x903, x904, x905, x906, x907, x908, x909,
                  x910, x911, x912, x913, x914, x915, x916, x917, x918, x919, x920, x921, x922,
                  x923, x924, x925, x926, x927, x928, x929, x930, x931, x932, x933, x934, x935,
                  x936, x937, x938, x939, x940, x941, x942, x943, x944, x945, x946, x947, x948,
                  x949, x950, x951, x952, x953, x954, x955, x956, x957},
      x958,
      LAYOUT_LOOKUP(layout2, _super));
  return x959;
}

} // namespace risc0::impl
