#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ OneHot_24_Struct exec_OneHot_24_(ExecContext& ctx,
                                            Val arg0,
                                            BoundLayout<OneHot_24_Layout> layout1) {
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:7)
  NondetRegStruct24Array x2 = map(
      Val24Array{Val(0),  Val(1),  Val(2),  Val(3),  Val(4),  Val(5),  Val(6),  Val(7),
                 Val(8),  Val(9),  Val(10), Val(11), Val(12), Val(13), Val(14), Val(15),
                 Val(16), Val(17), Val(18), Val(19), Val(20), Val(21), Val(22), Val(23)},
      LAYOUT_LOOKUP(layout1, _super),
      ([&](Val24Array::value_type x3, BoundLayout<NondetRegLayout24LayoutArray::value_type> x4) {
        NondetRegStruct x5 = exec_NondetBitReg(ctx, isz((x3 - arg0)), x4);
        return x5;
      }));
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:9)
  Val x6 = (x2[0]._super + x2[1]._super);
  Val x7 = ((x6 + x2[2]._super) + x2[3]._super);
  Val x8 = ((x7 + x2[4]._super) + x2[5]._super);
  Val x9 = ((x8 + x2[6]._super) + x2[7]._super);
  Val x10 = ((x9 + x2[8]._super) + x2[9]._super);
  Val x11 = ((x10 + x2[10]._super) + x2[11]._super);
  Val x12 = ((x11 + x2[12]._super) + x2[13]._super);
  Val x13 = ((x12 + x2[14]._super) + x2[15]._super);
  Val x14 = ((x13 + x2[16]._super) + x2[17]._super);
  Val x15 = ((x14 + x2[18]._super) + x2[19]._super);
  Val x16 = ((x15 + x2[20]._super) + x2[21]._super);
  Val x17 = ((x16 + x2[22]._super) + x2[23]._super);
  EQZ((x17 - Val(1)), "OneHot(zirgen/circuit/keccak2/one_hot.zir:9)");
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:11)
  Val x18 = (x2[2]._super * Val(2));
  Val x19 = (x2[3]._super * Val(3));
  Val x20 = (x2[4]._super * Val(4));
  Val x21 = (x2[5]._super * Val(5));
  Val x22 = (x2[6]._super * Val(6));
  Val x23 = (x2[7]._super * Val(7));
  Val x24 = (x2[8]._super * Val(8));
  Val x25 = (x2[9]._super * Val(9));
  Val x26 = (x2[10]._super * Val(10));
  Val x27 = (x2[11]._super * Val(11));
  Val x28 = (x2[12]._super * Val(12));
  Val x29 = (x2[13]._super * Val(13));
  Val x30 = (x2[14]._super * Val(14));
  Val x31 = (x2[15]._super * Val(15));
  Val x32 = (x2[16]._super * Val(16));
  Val x33 = (x2[17]._super * Val(17));
  Val x34 = (x2[18]._super * Val(18));
  Val x35 = (x2[19]._super * Val(19));
  Val x36 = (x2[20]._super * Val(20));
  Val x37 = (x2[21]._super * Val(21));
  Val x38 = (x2[22]._super * Val(22));
  Val x39 = (x2[23]._super * Val(23));
  Val x40 = (x2[1]._super + x18);
  Val x41 = (((x40 + x19) + x20) + x21);
  Val x42 = (((x41 + x22) + x23) + x24);
  Val x43 = (((x42 + x25) + x26) + x27);
  Val x44 = (((x43 + x28) + x29) + x30);
  Val x45 = (((x44 + x31) + x32) + x33);
  Val x46 = (((x45 + x34) + x35) + x36);
  Val x47 = (((x46 + x37) + x38) + x39);
  EQZ((x47 - arg0), "OneHot(zirgen/circuit/keccak2/one_hot.zir:11)");
  return OneHot_24_Struct{._super = x2};
}
__device__ UnpackReg_800__16_Struct exec_UnpackReg_800__16_(
    ExecContext& ctx, Val50Array arg0, BoundLayout<UnpackReg_800__16_Layout> layout1) {
  // Div(<preamble>:15)
  // UnpackNondet(zirgen/circuit/keccak2/pack.zir:42)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:47)
  DivStruct x2 = DivStruct{._super = bitAnd(arg0[0], Val(1))};
  // Div(<preamble>:19)
  Val x3 = (bitAnd(arg0[0], Val(2)) * Val(1006632961));
  Val x4 = (bitAnd(arg0[0], Val(4)) * Val(1509949441));
  Val x5 = (bitAnd(arg0[0], Val(8)) * Val(1761607681));
  Val x6 = (bitAnd(arg0[0], Val(16)) * Val(1887436801));
  Val x7 = (bitAnd(arg0[0], Val(32)) * Val(1950351361));
  Val x8 = (bitAnd(arg0[0], Val(64)) * Val(1981808641));
  Val x9 = (bitAnd(arg0[0], Val(128)) * Val(1997537281));
  Val x10 = (bitAnd(arg0[0], Val(256)) * Val(2005401601));
  Val x11 = (bitAnd(arg0[0], Val(512)) * Val(2009333761));
  Val x12 = (bitAnd(arg0[0], Val(1024)) * Val(2011299841));
  Val x13 = (bitAnd(arg0[0], Val(2048)) * Val(2012282881));
  Val x14 = (bitAnd(arg0[0], Val(4096)) * Val(2012774401));
  Val x15 = (bitAnd(arg0[0], Val(8192)) * Val(2013020161));
  Val x16 = (bitAnd(arg0[0], Val(16384)) * Val(2013143041));
  Val x17 = (bitAnd(arg0[0], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x18 = DivStruct{._super = bitAnd(arg0[1], Val(1))};
  // Div(<preamble>:19)
  Val x19 = (bitAnd(arg0[1], Val(2)) * Val(1006632961));
  Val x20 = (bitAnd(arg0[1], Val(4)) * Val(1509949441));
  Val x21 = (bitAnd(arg0[1], Val(8)) * Val(1761607681));
  Val x22 = (bitAnd(arg0[1], Val(16)) * Val(1887436801));
  Val x23 = (bitAnd(arg0[1], Val(32)) * Val(1950351361));
  Val x24 = (bitAnd(arg0[1], Val(64)) * Val(1981808641));
  Val x25 = (bitAnd(arg0[1], Val(128)) * Val(1997537281));
  Val x26 = (bitAnd(arg0[1], Val(256)) * Val(2005401601));
  Val x27 = (bitAnd(arg0[1], Val(512)) * Val(2009333761));
  Val x28 = (bitAnd(arg0[1], Val(1024)) * Val(2011299841));
  Val x29 = (bitAnd(arg0[1], Val(2048)) * Val(2012282881));
  Val x30 = (bitAnd(arg0[1], Val(4096)) * Val(2012774401));
  Val x31 = (bitAnd(arg0[1], Val(8192)) * Val(2013020161));
  Val x32 = (bitAnd(arg0[1], Val(16384)) * Val(2013143041));
  Val x33 = (bitAnd(arg0[1], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x34 = DivStruct{._super = bitAnd(arg0[2], Val(1))};
  // Div(<preamble>:19)
  Val x35 = (bitAnd(arg0[2], Val(2)) * Val(1006632961));
  Val x36 = (bitAnd(arg0[2], Val(4)) * Val(1509949441));
  Val x37 = (bitAnd(arg0[2], Val(8)) * Val(1761607681));
  Val x38 = (bitAnd(arg0[2], Val(16)) * Val(1887436801));
  Val x39 = (bitAnd(arg0[2], Val(32)) * Val(1950351361));
  Val x40 = (bitAnd(arg0[2], Val(64)) * Val(1981808641));
  Val x41 = (bitAnd(arg0[2], Val(128)) * Val(1997537281));
  Val x42 = (bitAnd(arg0[2], Val(256)) * Val(2005401601));
  Val x43 = (bitAnd(arg0[2], Val(512)) * Val(2009333761));
  Val x44 = (bitAnd(arg0[2], Val(1024)) * Val(2011299841));
  Val x45 = (bitAnd(arg0[2], Val(2048)) * Val(2012282881));
  Val x46 = (bitAnd(arg0[2], Val(4096)) * Val(2012774401));
  Val x47 = (bitAnd(arg0[2], Val(8192)) * Val(2013020161));
  Val x48 = (bitAnd(arg0[2], Val(16384)) * Val(2013143041));
  Val x49 = (bitAnd(arg0[2], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x50 = DivStruct{._super = bitAnd(arg0[3], Val(1))};
  // Div(<preamble>:19)
  Val x51 = (bitAnd(arg0[3], Val(2)) * Val(1006632961));
  Val x52 = (bitAnd(arg0[3], Val(4)) * Val(1509949441));
  Val x53 = (bitAnd(arg0[3], Val(8)) * Val(1761607681));
  Val x54 = (bitAnd(arg0[3], Val(16)) * Val(1887436801));
  Val x55 = (bitAnd(arg0[3], Val(32)) * Val(1950351361));
  Val x56 = (bitAnd(arg0[3], Val(64)) * Val(1981808641));
  Val x57 = (bitAnd(arg0[3], Val(128)) * Val(1997537281));
  Val x58 = (bitAnd(arg0[3], Val(256)) * Val(2005401601));
  Val x59 = (bitAnd(arg0[3], Val(512)) * Val(2009333761));
  Val x60 = (bitAnd(arg0[3], Val(1024)) * Val(2011299841));
  Val x61 = (bitAnd(arg0[3], Val(2048)) * Val(2012282881));
  Val x62 = (bitAnd(arg0[3], Val(4096)) * Val(2012774401));
  Val x63 = (bitAnd(arg0[3], Val(8192)) * Val(2013020161));
  Val x64 = (bitAnd(arg0[3], Val(16384)) * Val(2013143041));
  Val x65 = (bitAnd(arg0[3], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x66 = DivStruct{._super = bitAnd(arg0[4], Val(1))};
  // Div(<preamble>:19)
  Val x67 = (bitAnd(arg0[4], Val(2)) * Val(1006632961));
  Val x68 = (bitAnd(arg0[4], Val(4)) * Val(1509949441));
  Val x69 = (bitAnd(arg0[4], Val(8)) * Val(1761607681));
  Val x70 = (bitAnd(arg0[4], Val(16)) * Val(1887436801));
  Val x71 = (bitAnd(arg0[4], Val(32)) * Val(1950351361));
  Val x72 = (bitAnd(arg0[4], Val(64)) * Val(1981808641));
  Val x73 = (bitAnd(arg0[4], Val(128)) * Val(1997537281));
  Val x74 = (bitAnd(arg0[4], Val(256)) * Val(2005401601));
  Val x75 = (bitAnd(arg0[4], Val(512)) * Val(2009333761));
  Val x76 = (bitAnd(arg0[4], Val(1024)) * Val(2011299841));
  Val x77 = (bitAnd(arg0[4], Val(2048)) * Val(2012282881));
  Val x78 = (bitAnd(arg0[4], Val(4096)) * Val(2012774401));
  Val x79 = (bitAnd(arg0[4], Val(8192)) * Val(2013020161));
  Val x80 = (bitAnd(arg0[4], Val(16384)) * Val(2013143041));
  Val x81 = (bitAnd(arg0[4], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x82 = DivStruct{._super = bitAnd(arg0[5], Val(1))};
  // Div(<preamble>:19)
  Val x83 = (bitAnd(arg0[5], Val(2)) * Val(1006632961));
  Val x84 = (bitAnd(arg0[5], Val(4)) * Val(1509949441));
  Val x85 = (bitAnd(arg0[5], Val(8)) * Val(1761607681));
  Val x86 = (bitAnd(arg0[5], Val(16)) * Val(1887436801));
  Val x87 = (bitAnd(arg0[5], Val(32)) * Val(1950351361));
  Val x88 = (bitAnd(arg0[5], Val(64)) * Val(1981808641));
  Val x89 = (bitAnd(arg0[5], Val(128)) * Val(1997537281));
  Val x90 = (bitAnd(arg0[5], Val(256)) * Val(2005401601));
  Val x91 = (bitAnd(arg0[5], Val(512)) * Val(2009333761));
  Val x92 = (bitAnd(arg0[5], Val(1024)) * Val(2011299841));
  Val x93 = (bitAnd(arg0[5], Val(2048)) * Val(2012282881));
  Val x94 = (bitAnd(arg0[5], Val(4096)) * Val(2012774401));
  Val x95 = (bitAnd(arg0[5], Val(8192)) * Val(2013020161));
  Val x96 = (bitAnd(arg0[5], Val(16384)) * Val(2013143041));
  Val x97 = (bitAnd(arg0[5], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x98 = DivStruct{._super = bitAnd(arg0[6], Val(1))};
  // Div(<preamble>:19)
  Val x99 = (bitAnd(arg0[6], Val(2)) * Val(1006632961));
  Val x100 = (bitAnd(arg0[6], Val(4)) * Val(1509949441));
  Val x101 = (bitAnd(arg0[6], Val(8)) * Val(1761607681));
  Val x102 = (bitAnd(arg0[6], Val(16)) * Val(1887436801));
  Val x103 = (bitAnd(arg0[6], Val(32)) * Val(1950351361));
  Val x104 = (bitAnd(arg0[6], Val(64)) * Val(1981808641));
  Val x105 = (bitAnd(arg0[6], Val(128)) * Val(1997537281));
  Val x106 = (bitAnd(arg0[6], Val(256)) * Val(2005401601));
  Val x107 = (bitAnd(arg0[6], Val(512)) * Val(2009333761));
  Val x108 = (bitAnd(arg0[6], Val(1024)) * Val(2011299841));
  Val x109 = (bitAnd(arg0[6], Val(2048)) * Val(2012282881));
  Val x110 = (bitAnd(arg0[6], Val(4096)) * Val(2012774401));
  Val x111 = (bitAnd(arg0[6], Val(8192)) * Val(2013020161));
  Val x112 = (bitAnd(arg0[6], Val(16384)) * Val(2013143041));
  Val x113 = (bitAnd(arg0[6], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x114 = DivStruct{._super = bitAnd(arg0[7], Val(1))};
  // Div(<preamble>:19)
  Val x115 = (bitAnd(arg0[7], Val(2)) * Val(1006632961));
  Val x116 = (bitAnd(arg0[7], Val(4)) * Val(1509949441));
  Val x117 = (bitAnd(arg0[7], Val(8)) * Val(1761607681));
  Val x118 = (bitAnd(arg0[7], Val(16)) * Val(1887436801));
  Val x119 = (bitAnd(arg0[7], Val(32)) * Val(1950351361));
  Val x120 = (bitAnd(arg0[7], Val(64)) * Val(1981808641));
  Val x121 = (bitAnd(arg0[7], Val(128)) * Val(1997537281));
  Val x122 = (bitAnd(arg0[7], Val(256)) * Val(2005401601));
  Val x123 = (bitAnd(arg0[7], Val(512)) * Val(2009333761));
  Val x124 = (bitAnd(arg0[7], Val(1024)) * Val(2011299841));
  Val x125 = (bitAnd(arg0[7], Val(2048)) * Val(2012282881));
  Val x126 = (bitAnd(arg0[7], Val(4096)) * Val(2012774401));
  Val x127 = (bitAnd(arg0[7], Val(8192)) * Val(2013020161));
  Val x128 = (bitAnd(arg0[7], Val(16384)) * Val(2013143041));
  Val x129 = (bitAnd(arg0[7], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x130 = DivStruct{._super = bitAnd(arg0[8], Val(1))};
  // Div(<preamble>:19)
  Val x131 = (bitAnd(arg0[8], Val(2)) * Val(1006632961));
  Val x132 = (bitAnd(arg0[8], Val(4)) * Val(1509949441));
  Val x133 = (bitAnd(arg0[8], Val(8)) * Val(1761607681));
  Val x134 = (bitAnd(arg0[8], Val(16)) * Val(1887436801));
  Val x135 = (bitAnd(arg0[8], Val(32)) * Val(1950351361));
  Val x136 = (bitAnd(arg0[8], Val(64)) * Val(1981808641));
  Val x137 = (bitAnd(arg0[8], Val(128)) * Val(1997537281));
  Val x138 = (bitAnd(arg0[8], Val(256)) * Val(2005401601));
  Val x139 = (bitAnd(arg0[8], Val(512)) * Val(2009333761));
  Val x140 = (bitAnd(arg0[8], Val(1024)) * Val(2011299841));
  Val x141 = (bitAnd(arg0[8], Val(2048)) * Val(2012282881));
  Val x142 = (bitAnd(arg0[8], Val(4096)) * Val(2012774401));
  Val x143 = (bitAnd(arg0[8], Val(8192)) * Val(2013020161));
  Val x144 = (bitAnd(arg0[8], Val(16384)) * Val(2013143041));
  Val x145 = (bitAnd(arg0[8], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x146 = DivStruct{._super = bitAnd(arg0[9], Val(1))};
  // Div(<preamble>:19)
  Val x147 = (bitAnd(arg0[9], Val(2)) * Val(1006632961));
  Val x148 = (bitAnd(arg0[9], Val(4)) * Val(1509949441));
  Val x149 = (bitAnd(arg0[9], Val(8)) * Val(1761607681));
  Val x150 = (bitAnd(arg0[9], Val(16)) * Val(1887436801));
  Val x151 = (bitAnd(arg0[9], Val(32)) * Val(1950351361));
  Val x152 = (bitAnd(arg0[9], Val(64)) * Val(1981808641));
  Val x153 = (bitAnd(arg0[9], Val(128)) * Val(1997537281));
  Val x154 = (bitAnd(arg0[9], Val(256)) * Val(2005401601));
  Val x155 = (bitAnd(arg0[9], Val(512)) * Val(2009333761));
  Val x156 = (bitAnd(arg0[9], Val(1024)) * Val(2011299841));
  Val x157 = (bitAnd(arg0[9], Val(2048)) * Val(2012282881));
  Val x158 = (bitAnd(arg0[9], Val(4096)) * Val(2012774401));
  Val x159 = (bitAnd(arg0[9], Val(8192)) * Val(2013020161));
  Val x160 = (bitAnd(arg0[9], Val(16384)) * Val(2013143041));
  Val x161 = (bitAnd(arg0[9], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x162 = DivStruct{._super = bitAnd(arg0[10], Val(1))};
  // Div(<preamble>:19)
  Val x163 = (bitAnd(arg0[10], Val(2)) * Val(1006632961));
  Val x164 = (bitAnd(arg0[10], Val(4)) * Val(1509949441));
  Val x165 = (bitAnd(arg0[10], Val(8)) * Val(1761607681));
  Val x166 = (bitAnd(arg0[10], Val(16)) * Val(1887436801));
  Val x167 = (bitAnd(arg0[10], Val(32)) * Val(1950351361));
  Val x168 = (bitAnd(arg0[10], Val(64)) * Val(1981808641));
  Val x169 = (bitAnd(arg0[10], Val(128)) * Val(1997537281));
  Val x170 = (bitAnd(arg0[10], Val(256)) * Val(2005401601));
  Val x171 = (bitAnd(arg0[10], Val(512)) * Val(2009333761));
  Val x172 = (bitAnd(arg0[10], Val(1024)) * Val(2011299841));
  Val x173 = (bitAnd(arg0[10], Val(2048)) * Val(2012282881));
  Val x174 = (bitAnd(arg0[10], Val(4096)) * Val(2012774401));
  Val x175 = (bitAnd(arg0[10], Val(8192)) * Val(2013020161));
  Val x176 = (bitAnd(arg0[10], Val(16384)) * Val(2013143041));
  Val x177 = (bitAnd(arg0[10], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x178 = DivStruct{._super = bitAnd(arg0[11], Val(1))};
  // Div(<preamble>:19)
  Val x179 = (bitAnd(arg0[11], Val(2)) * Val(1006632961));
  Val x180 = (bitAnd(arg0[11], Val(4)) * Val(1509949441));
  Val x181 = (bitAnd(arg0[11], Val(8)) * Val(1761607681));
  Val x182 = (bitAnd(arg0[11], Val(16)) * Val(1887436801));
  Val x183 = (bitAnd(arg0[11], Val(32)) * Val(1950351361));
  Val x184 = (bitAnd(arg0[11], Val(64)) * Val(1981808641));
  Val x185 = (bitAnd(arg0[11], Val(128)) * Val(1997537281));
  Val x186 = (bitAnd(arg0[11], Val(256)) * Val(2005401601));
  Val x187 = (bitAnd(arg0[11], Val(512)) * Val(2009333761));
  Val x188 = (bitAnd(arg0[11], Val(1024)) * Val(2011299841));
  Val x189 = (bitAnd(arg0[11], Val(2048)) * Val(2012282881));
  Val x190 = (bitAnd(arg0[11], Val(4096)) * Val(2012774401));
  Val x191 = (bitAnd(arg0[11], Val(8192)) * Val(2013020161));
  Val x192 = (bitAnd(arg0[11], Val(16384)) * Val(2013143041));
  Val x193 = (bitAnd(arg0[11], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x194 = DivStruct{._super = bitAnd(arg0[12], Val(1))};
  // Div(<preamble>:19)
  Val x195 = (bitAnd(arg0[12], Val(2)) * Val(1006632961));
  Val x196 = (bitAnd(arg0[12], Val(4)) * Val(1509949441));
  Val x197 = (bitAnd(arg0[12], Val(8)) * Val(1761607681));
  Val x198 = (bitAnd(arg0[12], Val(16)) * Val(1887436801));
  Val x199 = (bitAnd(arg0[12], Val(32)) * Val(1950351361));
  Val x200 = (bitAnd(arg0[12], Val(64)) * Val(1981808641));
  Val x201 = (bitAnd(arg0[12], Val(128)) * Val(1997537281));
  Val x202 = (bitAnd(arg0[12], Val(256)) * Val(2005401601));
  Val x203 = (bitAnd(arg0[12], Val(512)) * Val(2009333761));
  Val x204 = (bitAnd(arg0[12], Val(1024)) * Val(2011299841));
  Val x205 = (bitAnd(arg0[12], Val(2048)) * Val(2012282881));
  Val x206 = (bitAnd(arg0[12], Val(4096)) * Val(2012774401));
  Val x207 = (bitAnd(arg0[12], Val(8192)) * Val(2013020161));
  Val x208 = (bitAnd(arg0[12], Val(16384)) * Val(2013143041));
  Val x209 = (bitAnd(arg0[12], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x210 = DivStruct{._super = bitAnd(arg0[13], Val(1))};
  // Div(<preamble>:19)
  Val x211 = (bitAnd(arg0[13], Val(2)) * Val(1006632961));
  Val x212 = (bitAnd(arg0[13], Val(4)) * Val(1509949441));
  Val x213 = (bitAnd(arg0[13], Val(8)) * Val(1761607681));
  Val x214 = (bitAnd(arg0[13], Val(16)) * Val(1887436801));
  Val x215 = (bitAnd(arg0[13], Val(32)) * Val(1950351361));
  Val x216 = (bitAnd(arg0[13], Val(64)) * Val(1981808641));
  Val x217 = (bitAnd(arg0[13], Val(128)) * Val(1997537281));
  Val x218 = (bitAnd(arg0[13], Val(256)) * Val(2005401601));
  Val x219 = (bitAnd(arg0[13], Val(512)) * Val(2009333761));
  Val x220 = (bitAnd(arg0[13], Val(1024)) * Val(2011299841));
  Val x221 = (bitAnd(arg0[13], Val(2048)) * Val(2012282881));
  Val x222 = (bitAnd(arg0[13], Val(4096)) * Val(2012774401));
  Val x223 = (bitAnd(arg0[13], Val(8192)) * Val(2013020161));
  Val x224 = (bitAnd(arg0[13], Val(16384)) * Val(2013143041));
  Val x225 = (bitAnd(arg0[13], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x226 = DivStruct{._super = bitAnd(arg0[14], Val(1))};
  // Div(<preamble>:19)
  Val x227 = (bitAnd(arg0[14], Val(2)) * Val(1006632961));
  Val x228 = (bitAnd(arg0[14], Val(4)) * Val(1509949441));
  Val x229 = (bitAnd(arg0[14], Val(8)) * Val(1761607681));
  Val x230 = (bitAnd(arg0[14], Val(16)) * Val(1887436801));
  Val x231 = (bitAnd(arg0[14], Val(32)) * Val(1950351361));
  Val x232 = (bitAnd(arg0[14], Val(64)) * Val(1981808641));
  Val x233 = (bitAnd(arg0[14], Val(128)) * Val(1997537281));
  Val x234 = (bitAnd(arg0[14], Val(256)) * Val(2005401601));
  Val x235 = (bitAnd(arg0[14], Val(512)) * Val(2009333761));
  Val x236 = (bitAnd(arg0[14], Val(1024)) * Val(2011299841));
  Val x237 = (bitAnd(arg0[14], Val(2048)) * Val(2012282881));
  Val x238 = (bitAnd(arg0[14], Val(4096)) * Val(2012774401));
  Val x239 = (bitAnd(arg0[14], Val(8192)) * Val(2013020161));
  Val x240 = (bitAnd(arg0[14], Val(16384)) * Val(2013143041));
  Val x241 = (bitAnd(arg0[14], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x242 = DivStruct{._super = bitAnd(arg0[15], Val(1))};
  // Div(<preamble>:19)
  Val x243 = (bitAnd(arg0[15], Val(2)) * Val(1006632961));
  Val x244 = (bitAnd(arg0[15], Val(4)) * Val(1509949441));
  Val x245 = (bitAnd(arg0[15], Val(8)) * Val(1761607681));
  Val x246 = (bitAnd(arg0[15], Val(16)) * Val(1887436801));
  Val x247 = (bitAnd(arg0[15], Val(32)) * Val(1950351361));
  Val x248 = (bitAnd(arg0[15], Val(64)) * Val(1981808641));
  Val x249 = (bitAnd(arg0[15], Val(128)) * Val(1997537281));
  Val x250 = (bitAnd(arg0[15], Val(256)) * Val(2005401601));
  Val x251 = (bitAnd(arg0[15], Val(512)) * Val(2009333761));
  Val x252 = (bitAnd(arg0[15], Val(1024)) * Val(2011299841));
  Val x253 = (bitAnd(arg0[15], Val(2048)) * Val(2012282881));
  Val x254 = (bitAnd(arg0[15], Val(4096)) * Val(2012774401));
  Val x255 = (bitAnd(arg0[15], Val(8192)) * Val(2013020161));
  Val x256 = (bitAnd(arg0[15], Val(16384)) * Val(2013143041));
  Val x257 = (bitAnd(arg0[15], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x258 = DivStruct{._super = bitAnd(arg0[16], Val(1))};
  // Div(<preamble>:19)
  Val x259 = (bitAnd(arg0[16], Val(2)) * Val(1006632961));
  Val x260 = (bitAnd(arg0[16], Val(4)) * Val(1509949441));
  Val x261 = (bitAnd(arg0[16], Val(8)) * Val(1761607681));
  Val x262 = (bitAnd(arg0[16], Val(16)) * Val(1887436801));
  Val x263 = (bitAnd(arg0[16], Val(32)) * Val(1950351361));
  Val x264 = (bitAnd(arg0[16], Val(64)) * Val(1981808641));
  Val x265 = (bitAnd(arg0[16], Val(128)) * Val(1997537281));
  Val x266 = (bitAnd(arg0[16], Val(256)) * Val(2005401601));
  Val x267 = (bitAnd(arg0[16], Val(512)) * Val(2009333761));
  Val x268 = (bitAnd(arg0[16], Val(1024)) * Val(2011299841));
  Val x269 = (bitAnd(arg0[16], Val(2048)) * Val(2012282881));
  Val x270 = (bitAnd(arg0[16], Val(4096)) * Val(2012774401));
  Val x271 = (bitAnd(arg0[16], Val(8192)) * Val(2013020161));
  Val x272 = (bitAnd(arg0[16], Val(16384)) * Val(2013143041));
  Val x273 = (bitAnd(arg0[16], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x274 = DivStruct{._super = bitAnd(arg0[17], Val(1))};
  // Div(<preamble>:19)
  Val x275 = (bitAnd(arg0[17], Val(2)) * Val(1006632961));
  Val x276 = (bitAnd(arg0[17], Val(4)) * Val(1509949441));
  Val x277 = (bitAnd(arg0[17], Val(8)) * Val(1761607681));
  Val x278 = (bitAnd(arg0[17], Val(16)) * Val(1887436801));
  Val x279 = (bitAnd(arg0[17], Val(32)) * Val(1950351361));
  Val x280 = (bitAnd(arg0[17], Val(64)) * Val(1981808641));
  Val x281 = (bitAnd(arg0[17], Val(128)) * Val(1997537281));
  Val x282 = (bitAnd(arg0[17], Val(256)) * Val(2005401601));
  Val x283 = (bitAnd(arg0[17], Val(512)) * Val(2009333761));
  Val x284 = (bitAnd(arg0[17], Val(1024)) * Val(2011299841));
  Val x285 = (bitAnd(arg0[17], Val(2048)) * Val(2012282881));
  Val x286 = (bitAnd(arg0[17], Val(4096)) * Val(2012774401));
  Val x287 = (bitAnd(arg0[17], Val(8192)) * Val(2013020161));
  Val x288 = (bitAnd(arg0[17], Val(16384)) * Val(2013143041));
  Val x289 = (bitAnd(arg0[17], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x290 = DivStruct{._super = bitAnd(arg0[18], Val(1))};
  // Div(<preamble>:19)
  Val x291 = (bitAnd(arg0[18], Val(2)) * Val(1006632961));
  Val x292 = (bitAnd(arg0[18], Val(4)) * Val(1509949441));
  Val x293 = (bitAnd(arg0[18], Val(8)) * Val(1761607681));
  Val x294 = (bitAnd(arg0[18], Val(16)) * Val(1887436801));
  Val x295 = (bitAnd(arg0[18], Val(32)) * Val(1950351361));
  Val x296 = (bitAnd(arg0[18], Val(64)) * Val(1981808641));
  Val x297 = (bitAnd(arg0[18], Val(128)) * Val(1997537281));
  Val x298 = (bitAnd(arg0[18], Val(256)) * Val(2005401601));
  Val x299 = (bitAnd(arg0[18], Val(512)) * Val(2009333761));
  Val x300 = (bitAnd(arg0[18], Val(1024)) * Val(2011299841));
  Val x301 = (bitAnd(arg0[18], Val(2048)) * Val(2012282881));
  Val x302 = (bitAnd(arg0[18], Val(4096)) * Val(2012774401));
  Val x303 = (bitAnd(arg0[18], Val(8192)) * Val(2013020161));
  Val x304 = (bitAnd(arg0[18], Val(16384)) * Val(2013143041));
  Val x305 = (bitAnd(arg0[18], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x306 = DivStruct{._super = bitAnd(arg0[19], Val(1))};
  // Div(<preamble>:19)
  Val x307 = (bitAnd(arg0[19], Val(2)) * Val(1006632961));
  Val x308 = (bitAnd(arg0[19], Val(4)) * Val(1509949441));
  Val x309 = (bitAnd(arg0[19], Val(8)) * Val(1761607681));
  Val x310 = (bitAnd(arg0[19], Val(16)) * Val(1887436801));
  Val x311 = (bitAnd(arg0[19], Val(32)) * Val(1950351361));
  Val x312 = (bitAnd(arg0[19], Val(64)) * Val(1981808641));
  Val x313 = (bitAnd(arg0[19], Val(128)) * Val(1997537281));
  Val x314 = (bitAnd(arg0[19], Val(256)) * Val(2005401601));
  Val x315 = (bitAnd(arg0[19], Val(512)) * Val(2009333761));
  Val x316 = (bitAnd(arg0[19], Val(1024)) * Val(2011299841));
  Val x317 = (bitAnd(arg0[19], Val(2048)) * Val(2012282881));
  Val x318 = (bitAnd(arg0[19], Val(4096)) * Val(2012774401));
  Val x319 = (bitAnd(arg0[19], Val(8192)) * Val(2013020161));
  Val x320 = (bitAnd(arg0[19], Val(16384)) * Val(2013143041));
  Val x321 = (bitAnd(arg0[19], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x322 = DivStruct{._super = bitAnd(arg0[20], Val(1))};
  // Div(<preamble>:19)
  Val x323 = (bitAnd(arg0[20], Val(2)) * Val(1006632961));
  Val x324 = (bitAnd(arg0[20], Val(4)) * Val(1509949441));
  Val x325 = (bitAnd(arg0[20], Val(8)) * Val(1761607681));
  Val x326 = (bitAnd(arg0[20], Val(16)) * Val(1887436801));
  Val x327 = (bitAnd(arg0[20], Val(32)) * Val(1950351361));
  Val x328 = (bitAnd(arg0[20], Val(64)) * Val(1981808641));
  Val x329 = (bitAnd(arg0[20], Val(128)) * Val(1997537281));
  Val x330 = (bitAnd(arg0[20], Val(256)) * Val(2005401601));
  Val x331 = (bitAnd(arg0[20], Val(512)) * Val(2009333761));
  Val x332 = (bitAnd(arg0[20], Val(1024)) * Val(2011299841));
  Val x333 = (bitAnd(arg0[20], Val(2048)) * Val(2012282881));
  Val x334 = (bitAnd(arg0[20], Val(4096)) * Val(2012774401));
  Val x335 = (bitAnd(arg0[20], Val(8192)) * Val(2013020161));
  Val x336 = (bitAnd(arg0[20], Val(16384)) * Val(2013143041));
  Val x337 = (bitAnd(arg0[20], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x338 = DivStruct{._super = bitAnd(arg0[21], Val(1))};
  // Div(<preamble>:19)
  Val x339 = (bitAnd(arg0[21], Val(2)) * Val(1006632961));
  Val x340 = (bitAnd(arg0[21], Val(4)) * Val(1509949441));
  Val x341 = (bitAnd(arg0[21], Val(8)) * Val(1761607681));
  Val x342 = (bitAnd(arg0[21], Val(16)) * Val(1887436801));
  Val x343 = (bitAnd(arg0[21], Val(32)) * Val(1950351361));
  Val x344 = (bitAnd(arg0[21], Val(64)) * Val(1981808641));
  Val x345 = (bitAnd(arg0[21], Val(128)) * Val(1997537281));
  Val x346 = (bitAnd(arg0[21], Val(256)) * Val(2005401601));
  Val x347 = (bitAnd(arg0[21], Val(512)) * Val(2009333761));
  Val x348 = (bitAnd(arg0[21], Val(1024)) * Val(2011299841));
  Val x349 = (bitAnd(arg0[21], Val(2048)) * Val(2012282881));
  Val x350 = (bitAnd(arg0[21], Val(4096)) * Val(2012774401));
  Val x351 = (bitAnd(arg0[21], Val(8192)) * Val(2013020161));
  Val x352 = (bitAnd(arg0[21], Val(16384)) * Val(2013143041));
  Val x353 = (bitAnd(arg0[21], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x354 = DivStruct{._super = bitAnd(arg0[22], Val(1))};
  // Div(<preamble>:19)
  Val x355 = (bitAnd(arg0[22], Val(2)) * Val(1006632961));
  Val x356 = (bitAnd(arg0[22], Val(4)) * Val(1509949441));
  Val x357 = (bitAnd(arg0[22], Val(8)) * Val(1761607681));
  Val x358 = (bitAnd(arg0[22], Val(16)) * Val(1887436801));
  Val x359 = (bitAnd(arg0[22], Val(32)) * Val(1950351361));
  Val x360 = (bitAnd(arg0[22], Val(64)) * Val(1981808641));
  Val x361 = (bitAnd(arg0[22], Val(128)) * Val(1997537281));
  Val x362 = (bitAnd(arg0[22], Val(256)) * Val(2005401601));
  Val x363 = (bitAnd(arg0[22], Val(512)) * Val(2009333761));
  Val x364 = (bitAnd(arg0[22], Val(1024)) * Val(2011299841));
  Val x365 = (bitAnd(arg0[22], Val(2048)) * Val(2012282881));
  Val x366 = (bitAnd(arg0[22], Val(4096)) * Val(2012774401));
  Val x367 = (bitAnd(arg0[22], Val(8192)) * Val(2013020161));
  Val x368 = (bitAnd(arg0[22], Val(16384)) * Val(2013143041));
  Val x369 = (bitAnd(arg0[22], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x370 = DivStruct{._super = bitAnd(arg0[23], Val(1))};
  // Div(<preamble>:19)
  Val x371 = (bitAnd(arg0[23], Val(2)) * Val(1006632961));
  Val x372 = (bitAnd(arg0[23], Val(4)) * Val(1509949441));
  Val x373 = (bitAnd(arg0[23], Val(8)) * Val(1761607681));
  Val x374 = (bitAnd(arg0[23], Val(16)) * Val(1887436801));
  Val x375 = (bitAnd(arg0[23], Val(32)) * Val(1950351361));
  Val x376 = (bitAnd(arg0[23], Val(64)) * Val(1981808641));
  Val x377 = (bitAnd(arg0[23], Val(128)) * Val(1997537281));
  Val x378 = (bitAnd(arg0[23], Val(256)) * Val(2005401601));
  Val x379 = (bitAnd(arg0[23], Val(512)) * Val(2009333761));
  Val x380 = (bitAnd(arg0[23], Val(1024)) * Val(2011299841));
  Val x381 = (bitAnd(arg0[23], Val(2048)) * Val(2012282881));
  Val x382 = (bitAnd(arg0[23], Val(4096)) * Val(2012774401));
  Val x383 = (bitAnd(arg0[23], Val(8192)) * Val(2013020161));
  Val x384 = (bitAnd(arg0[23], Val(16384)) * Val(2013143041));
  Val x385 = (bitAnd(arg0[23], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x386 = DivStruct{._super = bitAnd(arg0[24], Val(1))};
  // Div(<preamble>:19)
  Val x387 = (bitAnd(arg0[24], Val(2)) * Val(1006632961));
  Val x388 = (bitAnd(arg0[24], Val(4)) * Val(1509949441));
  Val x389 = (bitAnd(arg0[24], Val(8)) * Val(1761607681));
  Val x390 = (bitAnd(arg0[24], Val(16)) * Val(1887436801));
  Val x391 = (bitAnd(arg0[24], Val(32)) * Val(1950351361));
  Val x392 = (bitAnd(arg0[24], Val(64)) * Val(1981808641));
  Val x393 = (bitAnd(arg0[24], Val(128)) * Val(1997537281));
  Val x394 = (bitAnd(arg0[24], Val(256)) * Val(2005401601));
  Val x395 = (bitAnd(arg0[24], Val(512)) * Val(2009333761));
  Val x396 = (bitAnd(arg0[24], Val(1024)) * Val(2011299841));
  Val x397 = (bitAnd(arg0[24], Val(2048)) * Val(2012282881));
  Val x398 = (bitAnd(arg0[24], Val(4096)) * Val(2012774401));
  Val x399 = (bitAnd(arg0[24], Val(8192)) * Val(2013020161));
  Val x400 = (bitAnd(arg0[24], Val(16384)) * Val(2013143041));
  Val x401 = (bitAnd(arg0[24], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x402 = DivStruct{._super = bitAnd(arg0[25], Val(1))};
  // Div(<preamble>:19)
  Val x403 = (bitAnd(arg0[25], Val(2)) * Val(1006632961));
  Val x404 = (bitAnd(arg0[25], Val(4)) * Val(1509949441));
  Val x405 = (bitAnd(arg0[25], Val(8)) * Val(1761607681));
  Val x406 = (bitAnd(arg0[25], Val(16)) * Val(1887436801));
  Val x407 = (bitAnd(arg0[25], Val(32)) * Val(1950351361));
  Val x408 = (bitAnd(arg0[25], Val(64)) * Val(1981808641));
  Val x409 = (bitAnd(arg0[25], Val(128)) * Val(1997537281));
  Val x410 = (bitAnd(arg0[25], Val(256)) * Val(2005401601));
  Val x411 = (bitAnd(arg0[25], Val(512)) * Val(2009333761));
  Val x412 = (bitAnd(arg0[25], Val(1024)) * Val(2011299841));
  Val x413 = (bitAnd(arg0[25], Val(2048)) * Val(2012282881));
  Val x414 = (bitAnd(arg0[25], Val(4096)) * Val(2012774401));
  Val x415 = (bitAnd(arg0[25], Val(8192)) * Val(2013020161));
  Val x416 = (bitAnd(arg0[25], Val(16384)) * Val(2013143041));
  Val x417 = (bitAnd(arg0[25], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x418 = DivStruct{._super = bitAnd(arg0[26], Val(1))};
  // Div(<preamble>:19)
  Val x419 = (bitAnd(arg0[26], Val(2)) * Val(1006632961));
  Val x420 = (bitAnd(arg0[26], Val(4)) * Val(1509949441));
  Val x421 = (bitAnd(arg0[26], Val(8)) * Val(1761607681));
  Val x422 = (bitAnd(arg0[26], Val(16)) * Val(1887436801));
  Val x423 = (bitAnd(arg0[26], Val(32)) * Val(1950351361));
  Val x424 = (bitAnd(arg0[26], Val(64)) * Val(1981808641));
  Val x425 = (bitAnd(arg0[26], Val(128)) * Val(1997537281));
  Val x426 = (bitAnd(arg0[26], Val(256)) * Val(2005401601));
  Val x427 = (bitAnd(arg0[26], Val(512)) * Val(2009333761));
  Val x428 = (bitAnd(arg0[26], Val(1024)) * Val(2011299841));
  Val x429 = (bitAnd(arg0[26], Val(2048)) * Val(2012282881));
  Val x430 = (bitAnd(arg0[26], Val(4096)) * Val(2012774401));
  Val x431 = (bitAnd(arg0[26], Val(8192)) * Val(2013020161));
  Val x432 = (bitAnd(arg0[26], Val(16384)) * Val(2013143041));
  Val x433 = (bitAnd(arg0[26], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x434 = DivStruct{._super = bitAnd(arg0[27], Val(1))};
  // Div(<preamble>:19)
  Val x435 = (bitAnd(arg0[27], Val(2)) * Val(1006632961));
  Val x436 = (bitAnd(arg0[27], Val(4)) * Val(1509949441));
  Val x437 = (bitAnd(arg0[27], Val(8)) * Val(1761607681));
  Val x438 = (bitAnd(arg0[27], Val(16)) * Val(1887436801));
  Val x439 = (bitAnd(arg0[27], Val(32)) * Val(1950351361));
  Val x440 = (bitAnd(arg0[27], Val(64)) * Val(1981808641));
  Val x441 = (bitAnd(arg0[27], Val(128)) * Val(1997537281));
  Val x442 = (bitAnd(arg0[27], Val(256)) * Val(2005401601));
  Val x443 = (bitAnd(arg0[27], Val(512)) * Val(2009333761));
  Val x444 = (bitAnd(arg0[27], Val(1024)) * Val(2011299841));
  Val x445 = (bitAnd(arg0[27], Val(2048)) * Val(2012282881));
  Val x446 = (bitAnd(arg0[27], Val(4096)) * Val(2012774401));
  Val x447 = (bitAnd(arg0[27], Val(8192)) * Val(2013020161));
  Val x448 = (bitAnd(arg0[27], Val(16384)) * Val(2013143041));
  Val x449 = (bitAnd(arg0[27], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x450 = DivStruct{._super = bitAnd(arg0[28], Val(1))};
  // Div(<preamble>:19)
  Val x451 = (bitAnd(arg0[28], Val(2)) * Val(1006632961));
  Val x452 = (bitAnd(arg0[28], Val(4)) * Val(1509949441));
  Val x453 = (bitAnd(arg0[28], Val(8)) * Val(1761607681));
  Val x454 = (bitAnd(arg0[28], Val(16)) * Val(1887436801));
  Val x455 = (bitAnd(arg0[28], Val(32)) * Val(1950351361));
  Val x456 = (bitAnd(arg0[28], Val(64)) * Val(1981808641));
  Val x457 = (bitAnd(arg0[28], Val(128)) * Val(1997537281));
  Val x458 = (bitAnd(arg0[28], Val(256)) * Val(2005401601));
  Val x459 = (bitAnd(arg0[28], Val(512)) * Val(2009333761));
  Val x460 = (bitAnd(arg0[28], Val(1024)) * Val(2011299841));
  Val x461 = (bitAnd(arg0[28], Val(2048)) * Val(2012282881));
  Val x462 = (bitAnd(arg0[28], Val(4096)) * Val(2012774401));
  Val x463 = (bitAnd(arg0[28], Val(8192)) * Val(2013020161));
  Val x464 = (bitAnd(arg0[28], Val(16384)) * Val(2013143041));
  Val x465 = (bitAnd(arg0[28], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x466 = DivStruct{._super = bitAnd(arg0[29], Val(1))};
  // Div(<preamble>:19)
  Val x467 = (bitAnd(arg0[29], Val(2)) * Val(1006632961));
  Val x468 = (bitAnd(arg0[29], Val(4)) * Val(1509949441));
  Val x469 = (bitAnd(arg0[29], Val(8)) * Val(1761607681));
  Val x470 = (bitAnd(arg0[29], Val(16)) * Val(1887436801));
  Val x471 = (bitAnd(arg0[29], Val(32)) * Val(1950351361));
  Val x472 = (bitAnd(arg0[29], Val(64)) * Val(1981808641));
  Val x473 = (bitAnd(arg0[29], Val(128)) * Val(1997537281));
  Val x474 = (bitAnd(arg0[29], Val(256)) * Val(2005401601));
  Val x475 = (bitAnd(arg0[29], Val(512)) * Val(2009333761));
  Val x476 = (bitAnd(arg0[29], Val(1024)) * Val(2011299841));
  Val x477 = (bitAnd(arg0[29], Val(2048)) * Val(2012282881));
  Val x478 = (bitAnd(arg0[29], Val(4096)) * Val(2012774401));
  Val x479 = (bitAnd(arg0[29], Val(8192)) * Val(2013020161));
  Val x480 = (bitAnd(arg0[29], Val(16384)) * Val(2013143041));
  Val x481 = (bitAnd(arg0[29], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x482 = DivStruct{._super = bitAnd(arg0[30], Val(1))};
  // Div(<preamble>:19)
  Val x483 = (bitAnd(arg0[30], Val(2)) * Val(1006632961));
  Val x484 = (bitAnd(arg0[30], Val(4)) * Val(1509949441));
  Val x485 = (bitAnd(arg0[30], Val(8)) * Val(1761607681));
  Val x486 = (bitAnd(arg0[30], Val(16)) * Val(1887436801));
  Val x487 = (bitAnd(arg0[30], Val(32)) * Val(1950351361));
  Val x488 = (bitAnd(arg0[30], Val(64)) * Val(1981808641));
  Val x489 = (bitAnd(arg0[30], Val(128)) * Val(1997537281));
  Val x490 = (bitAnd(arg0[30], Val(256)) * Val(2005401601));
  Val x491 = (bitAnd(arg0[30], Val(512)) * Val(2009333761));
  Val x492 = (bitAnd(arg0[30], Val(1024)) * Val(2011299841));
  Val x493 = (bitAnd(arg0[30], Val(2048)) * Val(2012282881));
  Val x494 = (bitAnd(arg0[30], Val(4096)) * Val(2012774401));
  Val x495 = (bitAnd(arg0[30], Val(8192)) * Val(2013020161));
  Val x496 = (bitAnd(arg0[30], Val(16384)) * Val(2013143041));
  Val x497 = (bitAnd(arg0[30], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x498 = DivStruct{._super = bitAnd(arg0[31], Val(1))};
  // Div(<preamble>:19)
  Val x499 = (bitAnd(arg0[31], Val(2)) * Val(1006632961));
  Val x500 = (bitAnd(arg0[31], Val(4)) * Val(1509949441));
  Val x501 = (bitAnd(arg0[31], Val(8)) * Val(1761607681));
  Val x502 = (bitAnd(arg0[31], Val(16)) * Val(1887436801));
  Val x503 = (bitAnd(arg0[31], Val(32)) * Val(1950351361));
  Val x504 = (bitAnd(arg0[31], Val(64)) * Val(1981808641));
  Val x505 = (bitAnd(arg0[31], Val(128)) * Val(1997537281));
  Val x506 = (bitAnd(arg0[31], Val(256)) * Val(2005401601));
  Val x507 = (bitAnd(arg0[31], Val(512)) * Val(2009333761));
  Val x508 = (bitAnd(arg0[31], Val(1024)) * Val(2011299841));
  Val x509 = (bitAnd(arg0[31], Val(2048)) * Val(2012282881));
  Val x510 = (bitAnd(arg0[31], Val(4096)) * Val(2012774401));
  Val x511 = (bitAnd(arg0[31], Val(8192)) * Val(2013020161));
  Val x512 = (bitAnd(arg0[31], Val(16384)) * Val(2013143041));
  Val x513 = (bitAnd(arg0[31], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x514 = DivStruct{._super = bitAnd(arg0[32], Val(1))};
  // Div(<preamble>:19)
  Val x515 = (bitAnd(arg0[32], Val(2)) * Val(1006632961));
  Val x516 = (bitAnd(arg0[32], Val(4)) * Val(1509949441));
  Val x517 = (bitAnd(arg0[32], Val(8)) * Val(1761607681));
  Val x518 = (bitAnd(arg0[32], Val(16)) * Val(1887436801));
  Val x519 = (bitAnd(arg0[32], Val(32)) * Val(1950351361));
  Val x520 = (bitAnd(arg0[32], Val(64)) * Val(1981808641));
  Val x521 = (bitAnd(arg0[32], Val(128)) * Val(1997537281));
  Val x522 = (bitAnd(arg0[32], Val(256)) * Val(2005401601));
  Val x523 = (bitAnd(arg0[32], Val(512)) * Val(2009333761));
  Val x524 = (bitAnd(arg0[32], Val(1024)) * Val(2011299841));
  Val x525 = (bitAnd(arg0[32], Val(2048)) * Val(2012282881));
  Val x526 = (bitAnd(arg0[32], Val(4096)) * Val(2012774401));
  Val x527 = (bitAnd(arg0[32], Val(8192)) * Val(2013020161));
  Val x528 = (bitAnd(arg0[32], Val(16384)) * Val(2013143041));
  Val x529 = (bitAnd(arg0[32], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x530 = DivStruct{._super = bitAnd(arg0[33], Val(1))};
  // Div(<preamble>:19)
  Val x531 = (bitAnd(arg0[33], Val(2)) * Val(1006632961));
  Val x532 = (bitAnd(arg0[33], Val(4)) * Val(1509949441));
  Val x533 = (bitAnd(arg0[33], Val(8)) * Val(1761607681));
  Val x534 = (bitAnd(arg0[33], Val(16)) * Val(1887436801));
  Val x535 = (bitAnd(arg0[33], Val(32)) * Val(1950351361));
  Val x536 = (bitAnd(arg0[33], Val(64)) * Val(1981808641));
  Val x537 = (bitAnd(arg0[33], Val(128)) * Val(1997537281));
  Val x538 = (bitAnd(arg0[33], Val(256)) * Val(2005401601));
  Val x539 = (bitAnd(arg0[33], Val(512)) * Val(2009333761));
  Val x540 = (bitAnd(arg0[33], Val(1024)) * Val(2011299841));
  Val x541 = (bitAnd(arg0[33], Val(2048)) * Val(2012282881));
  Val x542 = (bitAnd(arg0[33], Val(4096)) * Val(2012774401));
  Val x543 = (bitAnd(arg0[33], Val(8192)) * Val(2013020161));
  Val x544 = (bitAnd(arg0[33], Val(16384)) * Val(2013143041));
  Val x545 = (bitAnd(arg0[33], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x546 = DivStruct{._super = bitAnd(arg0[34], Val(1))};
  // Div(<preamble>:19)
  Val x547 = (bitAnd(arg0[34], Val(2)) * Val(1006632961));
  Val x548 = (bitAnd(arg0[34], Val(4)) * Val(1509949441));
  Val x549 = (bitAnd(arg0[34], Val(8)) * Val(1761607681));
  Val x550 = (bitAnd(arg0[34], Val(16)) * Val(1887436801));
  Val x551 = (bitAnd(arg0[34], Val(32)) * Val(1950351361));
  Val x552 = (bitAnd(arg0[34], Val(64)) * Val(1981808641));
  Val x553 = (bitAnd(arg0[34], Val(128)) * Val(1997537281));
  Val x554 = (bitAnd(arg0[34], Val(256)) * Val(2005401601));
  Val x555 = (bitAnd(arg0[34], Val(512)) * Val(2009333761));
  Val x556 = (bitAnd(arg0[34], Val(1024)) * Val(2011299841));
  Val x557 = (bitAnd(arg0[34], Val(2048)) * Val(2012282881));
  Val x558 = (bitAnd(arg0[34], Val(4096)) * Val(2012774401));
  Val x559 = (bitAnd(arg0[34], Val(8192)) * Val(2013020161));
  Val x560 = (bitAnd(arg0[34], Val(16384)) * Val(2013143041));
  Val x561 = (bitAnd(arg0[34], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x562 = DivStruct{._super = bitAnd(arg0[35], Val(1))};
  // Div(<preamble>:19)
  Val x563 = (bitAnd(arg0[35], Val(2)) * Val(1006632961));
  Val x564 = (bitAnd(arg0[35], Val(4)) * Val(1509949441));
  Val x565 = (bitAnd(arg0[35], Val(8)) * Val(1761607681));
  Val x566 = (bitAnd(arg0[35], Val(16)) * Val(1887436801));
  Val x567 = (bitAnd(arg0[35], Val(32)) * Val(1950351361));
  Val x568 = (bitAnd(arg0[35], Val(64)) * Val(1981808641));
  Val x569 = (bitAnd(arg0[35], Val(128)) * Val(1997537281));
  Val x570 = (bitAnd(arg0[35], Val(256)) * Val(2005401601));
  Val x571 = (bitAnd(arg0[35], Val(512)) * Val(2009333761));
  Val x572 = (bitAnd(arg0[35], Val(1024)) * Val(2011299841));
  Val x573 = (bitAnd(arg0[35], Val(2048)) * Val(2012282881));
  Val x574 = (bitAnd(arg0[35], Val(4096)) * Val(2012774401));
  Val x575 = (bitAnd(arg0[35], Val(8192)) * Val(2013020161));
  Val x576 = (bitAnd(arg0[35], Val(16384)) * Val(2013143041));
  Val x577 = (bitAnd(arg0[35], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x578 = DivStruct{._super = bitAnd(arg0[36], Val(1))};
  // Div(<preamble>:19)
  Val x579 = (bitAnd(arg0[36], Val(2)) * Val(1006632961));
  Val x580 = (bitAnd(arg0[36], Val(4)) * Val(1509949441));
  Val x581 = (bitAnd(arg0[36], Val(8)) * Val(1761607681));
  Val x582 = (bitAnd(arg0[36], Val(16)) * Val(1887436801));
  Val x583 = (bitAnd(arg0[36], Val(32)) * Val(1950351361));
  Val x584 = (bitAnd(arg0[36], Val(64)) * Val(1981808641));
  Val x585 = (bitAnd(arg0[36], Val(128)) * Val(1997537281));
  Val x586 = (bitAnd(arg0[36], Val(256)) * Val(2005401601));
  Val x587 = (bitAnd(arg0[36], Val(512)) * Val(2009333761));
  Val x588 = (bitAnd(arg0[36], Val(1024)) * Val(2011299841));
  Val x589 = (bitAnd(arg0[36], Val(2048)) * Val(2012282881));
  Val x590 = (bitAnd(arg0[36], Val(4096)) * Val(2012774401));
  Val x591 = (bitAnd(arg0[36], Val(8192)) * Val(2013020161));
  Val x592 = (bitAnd(arg0[36], Val(16384)) * Val(2013143041));
  Val x593 = (bitAnd(arg0[36], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x594 = DivStruct{._super = bitAnd(arg0[37], Val(1))};
  // Div(<preamble>:19)
  Val x595 = (bitAnd(arg0[37], Val(2)) * Val(1006632961));
  Val x596 = (bitAnd(arg0[37], Val(4)) * Val(1509949441));
  Val x597 = (bitAnd(arg0[37], Val(8)) * Val(1761607681));
  Val x598 = (bitAnd(arg0[37], Val(16)) * Val(1887436801));
  Val x599 = (bitAnd(arg0[37], Val(32)) * Val(1950351361));
  Val x600 = (bitAnd(arg0[37], Val(64)) * Val(1981808641));
  Val x601 = (bitAnd(arg0[37], Val(128)) * Val(1997537281));
  Val x602 = (bitAnd(arg0[37], Val(256)) * Val(2005401601));
  Val x603 = (bitAnd(arg0[37], Val(512)) * Val(2009333761));
  Val x604 = (bitAnd(arg0[37], Val(1024)) * Val(2011299841));
  Val x605 = (bitAnd(arg0[37], Val(2048)) * Val(2012282881));
  Val x606 = (bitAnd(arg0[37], Val(4096)) * Val(2012774401));
  Val x607 = (bitAnd(arg0[37], Val(8192)) * Val(2013020161));
  Val x608 = (bitAnd(arg0[37], Val(16384)) * Val(2013143041));
  Val x609 = (bitAnd(arg0[37], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x610 = DivStruct{._super = bitAnd(arg0[38], Val(1))};
  // Div(<preamble>:19)
  Val x611 = (bitAnd(arg0[38], Val(2)) * Val(1006632961));
  Val x612 = (bitAnd(arg0[38], Val(4)) * Val(1509949441));
  Val x613 = (bitAnd(arg0[38], Val(8)) * Val(1761607681));
  Val x614 = (bitAnd(arg0[38], Val(16)) * Val(1887436801));
  Val x615 = (bitAnd(arg0[38], Val(32)) * Val(1950351361));
  Val x616 = (bitAnd(arg0[38], Val(64)) * Val(1981808641));
  Val x617 = (bitAnd(arg0[38], Val(128)) * Val(1997537281));
  Val x618 = (bitAnd(arg0[38], Val(256)) * Val(2005401601));
  Val x619 = (bitAnd(arg0[38], Val(512)) * Val(2009333761));
  Val x620 = (bitAnd(arg0[38], Val(1024)) * Val(2011299841));
  Val x621 = (bitAnd(arg0[38], Val(2048)) * Val(2012282881));
  Val x622 = (bitAnd(arg0[38], Val(4096)) * Val(2012774401));
  Val x623 = (bitAnd(arg0[38], Val(8192)) * Val(2013020161));
  Val x624 = (bitAnd(arg0[38], Val(16384)) * Val(2013143041));
  Val x625 = (bitAnd(arg0[38], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x626 = DivStruct{._super = bitAnd(arg0[39], Val(1))};
  // Div(<preamble>:19)
  Val x627 = (bitAnd(arg0[39], Val(2)) * Val(1006632961));
  Val x628 = (bitAnd(arg0[39], Val(4)) * Val(1509949441));
  Val x629 = (bitAnd(arg0[39], Val(8)) * Val(1761607681));
  Val x630 = (bitAnd(arg0[39], Val(16)) * Val(1887436801));
  Val x631 = (bitAnd(arg0[39], Val(32)) * Val(1950351361));
  Val x632 = (bitAnd(arg0[39], Val(64)) * Val(1981808641));
  Val x633 = (bitAnd(arg0[39], Val(128)) * Val(1997537281));
  Val x634 = (bitAnd(arg0[39], Val(256)) * Val(2005401601));
  Val x635 = (bitAnd(arg0[39], Val(512)) * Val(2009333761));
  Val x636 = (bitAnd(arg0[39], Val(1024)) * Val(2011299841));
  Val x637 = (bitAnd(arg0[39], Val(2048)) * Val(2012282881));
  Val x638 = (bitAnd(arg0[39], Val(4096)) * Val(2012774401));
  Val x639 = (bitAnd(arg0[39], Val(8192)) * Val(2013020161));
  Val x640 = (bitAnd(arg0[39], Val(16384)) * Val(2013143041));
  Val x641 = (bitAnd(arg0[39], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x642 = DivStruct{._super = bitAnd(arg0[40], Val(1))};
  // Div(<preamble>:19)
  Val x643 = (bitAnd(arg0[40], Val(2)) * Val(1006632961));
  Val x644 = (bitAnd(arg0[40], Val(4)) * Val(1509949441));
  Val x645 = (bitAnd(arg0[40], Val(8)) * Val(1761607681));
  Val x646 = (bitAnd(arg0[40], Val(16)) * Val(1887436801));
  Val x647 = (bitAnd(arg0[40], Val(32)) * Val(1950351361));
  Val x648 = (bitAnd(arg0[40], Val(64)) * Val(1981808641));
  Val x649 = (bitAnd(arg0[40], Val(128)) * Val(1997537281));
  Val x650 = (bitAnd(arg0[40], Val(256)) * Val(2005401601));
  Val x651 = (bitAnd(arg0[40], Val(512)) * Val(2009333761));
  Val x652 = (bitAnd(arg0[40], Val(1024)) * Val(2011299841));
  Val x653 = (bitAnd(arg0[40], Val(2048)) * Val(2012282881));
  Val x654 = (bitAnd(arg0[40], Val(4096)) * Val(2012774401));
  Val x655 = (bitAnd(arg0[40], Val(8192)) * Val(2013020161));
  Val x656 = (bitAnd(arg0[40], Val(16384)) * Val(2013143041));
  Val x657 = (bitAnd(arg0[40], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x658 = DivStruct{._super = bitAnd(arg0[41], Val(1))};
  // Div(<preamble>:19)
  Val x659 = (bitAnd(arg0[41], Val(2)) * Val(1006632961));
  Val x660 = (bitAnd(arg0[41], Val(4)) * Val(1509949441));
  Val x661 = (bitAnd(arg0[41], Val(8)) * Val(1761607681));
  Val x662 = (bitAnd(arg0[41], Val(16)) * Val(1887436801));
  Val x663 = (bitAnd(arg0[41], Val(32)) * Val(1950351361));
  Val x664 = (bitAnd(arg0[41], Val(64)) * Val(1981808641));
  Val x665 = (bitAnd(arg0[41], Val(128)) * Val(1997537281));
  Val x666 = (bitAnd(arg0[41], Val(256)) * Val(2005401601));
  Val x667 = (bitAnd(arg0[41], Val(512)) * Val(2009333761));
  Val x668 = (bitAnd(arg0[41], Val(1024)) * Val(2011299841));
  Val x669 = (bitAnd(arg0[41], Val(2048)) * Val(2012282881));
  Val x670 = (bitAnd(arg0[41], Val(4096)) * Val(2012774401));
  Val x671 = (bitAnd(arg0[41], Val(8192)) * Val(2013020161));
  Val x672 = (bitAnd(arg0[41], Val(16384)) * Val(2013143041));
  Val x673 = (bitAnd(arg0[41], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x674 = DivStruct{._super = bitAnd(arg0[42], Val(1))};
  // Div(<preamble>:19)
  Val x675 = (bitAnd(arg0[42], Val(2)) * Val(1006632961));
  Val x676 = (bitAnd(arg0[42], Val(4)) * Val(1509949441));
  Val x677 = (bitAnd(arg0[42], Val(8)) * Val(1761607681));
  Val x678 = (bitAnd(arg0[42], Val(16)) * Val(1887436801));
  Val x679 = (bitAnd(arg0[42], Val(32)) * Val(1950351361));
  Val x680 = (bitAnd(arg0[42], Val(64)) * Val(1981808641));
  Val x681 = (bitAnd(arg0[42], Val(128)) * Val(1997537281));
  Val x682 = (bitAnd(arg0[42], Val(256)) * Val(2005401601));
  Val x683 = (bitAnd(arg0[42], Val(512)) * Val(2009333761));
  Val x684 = (bitAnd(arg0[42], Val(1024)) * Val(2011299841));
  Val x685 = (bitAnd(arg0[42], Val(2048)) * Val(2012282881));
  Val x686 = (bitAnd(arg0[42], Val(4096)) * Val(2012774401));
  Val x687 = (bitAnd(arg0[42], Val(8192)) * Val(2013020161));
  Val x688 = (bitAnd(arg0[42], Val(16384)) * Val(2013143041));
  Val x689 = (bitAnd(arg0[42], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x690 = DivStruct{._super = bitAnd(arg0[43], Val(1))};
  // Div(<preamble>:19)
  Val x691 = (bitAnd(arg0[43], Val(2)) * Val(1006632961));
  Val x692 = (bitAnd(arg0[43], Val(4)) * Val(1509949441));
  Val x693 = (bitAnd(arg0[43], Val(8)) * Val(1761607681));
  Val x694 = (bitAnd(arg0[43], Val(16)) * Val(1887436801));
  Val x695 = (bitAnd(arg0[43], Val(32)) * Val(1950351361));
  Val x696 = (bitAnd(arg0[43], Val(64)) * Val(1981808641));
  Val x697 = (bitAnd(arg0[43], Val(128)) * Val(1997537281));
  Val x698 = (bitAnd(arg0[43], Val(256)) * Val(2005401601));
  Val x699 = (bitAnd(arg0[43], Val(512)) * Val(2009333761));
  Val x700 = (bitAnd(arg0[43], Val(1024)) * Val(2011299841));
  Val x701 = (bitAnd(arg0[43], Val(2048)) * Val(2012282881));
  Val x702 = (bitAnd(arg0[43], Val(4096)) * Val(2012774401));
  Val x703 = (bitAnd(arg0[43], Val(8192)) * Val(2013020161));
  Val x704 = (bitAnd(arg0[43], Val(16384)) * Val(2013143041));
  Val x705 = (bitAnd(arg0[43], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x706 = DivStruct{._super = bitAnd(arg0[44], Val(1))};
  // Div(<preamble>:19)
  Val x707 = (bitAnd(arg0[44], Val(2)) * Val(1006632961));
  Val x708 = (bitAnd(arg0[44], Val(4)) * Val(1509949441));
  Val x709 = (bitAnd(arg0[44], Val(8)) * Val(1761607681));
  Val x710 = (bitAnd(arg0[44], Val(16)) * Val(1887436801));
  Val x711 = (bitAnd(arg0[44], Val(32)) * Val(1950351361));
  Val x712 = (bitAnd(arg0[44], Val(64)) * Val(1981808641));
  Val x713 = (bitAnd(arg0[44], Val(128)) * Val(1997537281));
  Val x714 = (bitAnd(arg0[44], Val(256)) * Val(2005401601));
  Val x715 = (bitAnd(arg0[44], Val(512)) * Val(2009333761));
  Val x716 = (bitAnd(arg0[44], Val(1024)) * Val(2011299841));
  Val x717 = (bitAnd(arg0[44], Val(2048)) * Val(2012282881));
  Val x718 = (bitAnd(arg0[44], Val(4096)) * Val(2012774401));
  Val x719 = (bitAnd(arg0[44], Val(8192)) * Val(2013020161));
  Val x720 = (bitAnd(arg0[44], Val(16384)) * Val(2013143041));
  Val x721 = (bitAnd(arg0[44], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x722 = DivStruct{._super = bitAnd(arg0[45], Val(1))};
  // Div(<preamble>:19)
  Val x723 = (bitAnd(arg0[45], Val(2)) * Val(1006632961));
  Val x724 = (bitAnd(arg0[45], Val(4)) * Val(1509949441));
  Val x725 = (bitAnd(arg0[45], Val(8)) * Val(1761607681));
  Val x726 = (bitAnd(arg0[45], Val(16)) * Val(1887436801));
  Val x727 = (bitAnd(arg0[45], Val(32)) * Val(1950351361));
  Val x728 = (bitAnd(arg0[45], Val(64)) * Val(1981808641));
  Val x729 = (bitAnd(arg0[45], Val(128)) * Val(1997537281));
  Val x730 = (bitAnd(arg0[45], Val(256)) * Val(2005401601));
  Val x731 = (bitAnd(arg0[45], Val(512)) * Val(2009333761));
  Val x732 = (bitAnd(arg0[45], Val(1024)) * Val(2011299841));
  Val x733 = (bitAnd(arg0[45], Val(2048)) * Val(2012282881));
  Val x734 = (bitAnd(arg0[45], Val(4096)) * Val(2012774401));
  Val x735 = (bitAnd(arg0[45], Val(8192)) * Val(2013020161));
  Val x736 = (bitAnd(arg0[45], Val(16384)) * Val(2013143041));
  Val x737 = (bitAnd(arg0[45], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x738 = DivStruct{._super = bitAnd(arg0[46], Val(1))};
  // Div(<preamble>:19)
  Val x739 = (bitAnd(arg0[46], Val(2)) * Val(1006632961));
  Val x740 = (bitAnd(arg0[46], Val(4)) * Val(1509949441));
  Val x741 = (bitAnd(arg0[46], Val(8)) * Val(1761607681));
  Val x742 = (bitAnd(arg0[46], Val(16)) * Val(1887436801));
  Val x743 = (bitAnd(arg0[46], Val(32)) * Val(1950351361));
  Val x744 = (bitAnd(arg0[46], Val(64)) * Val(1981808641));
  Val x745 = (bitAnd(arg0[46], Val(128)) * Val(1997537281));
  Val x746 = (bitAnd(arg0[46], Val(256)) * Val(2005401601));
  Val x747 = (bitAnd(arg0[46], Val(512)) * Val(2009333761));
  Val x748 = (bitAnd(arg0[46], Val(1024)) * Val(2011299841));
  Val x749 = (bitAnd(arg0[46], Val(2048)) * Val(2012282881));
  Val x750 = (bitAnd(arg0[46], Val(4096)) * Val(2012774401));
  Val x751 = (bitAnd(arg0[46], Val(8192)) * Val(2013020161));
  Val x752 = (bitAnd(arg0[46], Val(16384)) * Val(2013143041));
  Val x753 = (bitAnd(arg0[46], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x754 = DivStruct{._super = bitAnd(arg0[47], Val(1))};
  // Div(<preamble>:19)
  Val x755 = (bitAnd(arg0[47], Val(2)) * Val(1006632961));
  Val x756 = (bitAnd(arg0[47], Val(4)) * Val(1509949441));
  Val x757 = (bitAnd(arg0[47], Val(8)) * Val(1761607681));
  Val x758 = (bitAnd(arg0[47], Val(16)) * Val(1887436801));
  Val x759 = (bitAnd(arg0[47], Val(32)) * Val(1950351361));
  Val x760 = (bitAnd(arg0[47], Val(64)) * Val(1981808641));
  Val x761 = (bitAnd(arg0[47], Val(128)) * Val(1997537281));
  Val x762 = (bitAnd(arg0[47], Val(256)) * Val(2005401601));
  Val x763 = (bitAnd(arg0[47], Val(512)) * Val(2009333761));
  Val x764 = (bitAnd(arg0[47], Val(1024)) * Val(2011299841));
  Val x765 = (bitAnd(arg0[47], Val(2048)) * Val(2012282881));
  Val x766 = (bitAnd(arg0[47], Val(4096)) * Val(2012774401));
  Val x767 = (bitAnd(arg0[47], Val(8192)) * Val(2013020161));
  Val x768 = (bitAnd(arg0[47], Val(16384)) * Val(2013143041));
  Val x769 = (bitAnd(arg0[47], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x770 = DivStruct{._super = bitAnd(arg0[48], Val(1))};
  // Div(<preamble>:19)
  Val x771 = (bitAnd(arg0[48], Val(2)) * Val(1006632961));
  Val x772 = (bitAnd(arg0[48], Val(4)) * Val(1509949441));
  Val x773 = (bitAnd(arg0[48], Val(8)) * Val(1761607681));
  Val x774 = (bitAnd(arg0[48], Val(16)) * Val(1887436801));
  Val x775 = (bitAnd(arg0[48], Val(32)) * Val(1950351361));
  Val x776 = (bitAnd(arg0[48], Val(64)) * Val(1981808641));
  Val x777 = (bitAnd(arg0[48], Val(128)) * Val(1997537281));
  Val x778 = (bitAnd(arg0[48], Val(256)) * Val(2005401601));
  Val x779 = (bitAnd(arg0[48], Val(512)) * Val(2009333761));
  Val x780 = (bitAnd(arg0[48], Val(1024)) * Val(2011299841));
  Val x781 = (bitAnd(arg0[48], Val(2048)) * Val(2012282881));
  Val x782 = (bitAnd(arg0[48], Val(4096)) * Val(2012774401));
  Val x783 = (bitAnd(arg0[48], Val(8192)) * Val(2013020161));
  Val x784 = (bitAnd(arg0[48], Val(16384)) * Val(2013143041));
  Val x785 = (bitAnd(arg0[48], Val(32768)) * Val(2013204481));
  // Div(<preamble>:15)
  DivStruct x786 = DivStruct{._super = bitAnd(arg0[49], Val(1))};
  // Div(<preamble>:19)
  Val x787 = (bitAnd(arg0[49], Val(2)) * Val(1006632961));
  Val x788 = (bitAnd(arg0[49], Val(4)) * Val(1509949441));
  Val x789 = (bitAnd(arg0[49], Val(8)) * Val(1761607681));
  Val x790 = (bitAnd(arg0[49], Val(16)) * Val(1887436801));
  Val x791 = (bitAnd(arg0[49], Val(32)) * Val(1950351361));
  Val x792 = (bitAnd(arg0[49], Val(64)) * Val(1981808641));
  Val x793 = (bitAnd(arg0[49], Val(128)) * Val(1997537281));
  Val x794 = (bitAnd(arg0[49], Val(256)) * Val(2005401601));
  Val x795 = (bitAnd(arg0[49], Val(512)) * Val(2009333761));
  Val x796 = (bitAnd(arg0[49], Val(1024)) * Val(2011299841));
  Val x797 = (bitAnd(arg0[49], Val(2048)) * Val(2012282881));
  Val x798 = (bitAnd(arg0[49], Val(4096)) * Val(2012774401));
  Val x799 = (bitAnd(arg0[49], Val(8192)) * Val(2013020161));
  Val x800 = (bitAnd(arg0[49], Val(16384)) * Val(2013143041));
  Val x801 = (bitAnd(arg0[49], Val(32768)) * Val(2013204481));
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  UnpackReg_800__16__SuperStruct800Array x802 = map(
      Val800Array{
          Val(0),   Val(1),   Val(2),   Val(3),   Val(4),   Val(5),   Val(6),   Val(7),   Val(8),
          Val(9),   Val(10),  Val(11),  Val(12),  Val(13),  Val(14),  Val(15),  Val(16),  Val(17),
          Val(18),  Val(19),  Val(20),  Val(21),  Val(22),  Val(23),  Val(24),  Val(25),  Val(26),
          Val(27),  Val(28),  Val(29),  Val(30),  Val(31),  Val(32),  Val(33),  Val(34),  Val(35),
          Val(36),  Val(37),  Val(38),  Val(39),  Val(40),  Val(41),  Val(42),  Val(43),  Val(44),
          Val(45),  Val(46),  Val(47),  Val(48),  Val(49),  Val(50),  Val(51),  Val(52),  Val(53),
          Val(54),  Val(55),  Val(56),  Val(57),  Val(58),  Val(59),  Val(60),  Val(61),  Val(62),
          Val(63),  Val(64),  Val(65),  Val(66),  Val(67),  Val(68),  Val(69),  Val(70),  Val(71),
          Val(72),  Val(73),  Val(74),  Val(75),  Val(76),  Val(77),  Val(78),  Val(79),  Val(80),
          Val(81),  Val(82),  Val(83),  Val(84),  Val(85),  Val(86),  Val(87),  Val(88),  Val(89),
          Val(90),  Val(91),  Val(92),  Val(93),  Val(94),  Val(95),  Val(96),  Val(97),  Val(98),
          Val(99),  Val(100), Val(101), Val(102), Val(103), Val(104), Val(105), Val(106), Val(107),
          Val(108), Val(109), Val(110), Val(111), Val(112), Val(113), Val(114), Val(115), Val(116),
          Val(117), Val(118), Val(119), Val(120), Val(121), Val(122), Val(123), Val(124), Val(125),
          Val(126), Val(127), Val(128), Val(129), Val(130), Val(131), Val(132), Val(133), Val(134),
          Val(135), Val(136), Val(137), Val(138), Val(139), Val(140), Val(141), Val(142), Val(143),
          Val(144), Val(145), Val(146), Val(147), Val(148), Val(149), Val(150), Val(151), Val(152),
          Val(153), Val(154), Val(155), Val(156), Val(157), Val(158), Val(159), Val(160), Val(161),
          Val(162), Val(163), Val(164), Val(165), Val(166), Val(167), Val(168), Val(169), Val(170),
          Val(171), Val(172), Val(173), Val(174), Val(175), Val(176), Val(177), Val(178), Val(179),
          Val(180), Val(181), Val(182), Val(183), Val(184), Val(185), Val(186), Val(187), Val(188),
          Val(189), Val(190), Val(191), Val(192), Val(193), Val(194), Val(195), Val(196), Val(197),
          Val(198), Val(199), Val(200), Val(201), Val(202), Val(203), Val(204), Val(205), Val(206),
          Val(207), Val(208), Val(209), Val(210), Val(211), Val(212), Val(213), Val(214), Val(215),
          Val(216), Val(217), Val(218), Val(219), Val(220), Val(221), Val(222), Val(223), Val(224),
          Val(225), Val(226), Val(227), Val(228), Val(229), Val(230), Val(231), Val(232), Val(233),
          Val(234), Val(235), Val(236), Val(237), Val(238), Val(239), Val(240), Val(241), Val(242),
          Val(243), Val(244), Val(245), Val(246), Val(247), Val(248), Val(249), Val(250), Val(251),
          Val(252), Val(253), Val(254), Val(255), Val(256), Val(257), Val(258), Val(259), Val(260),
          Val(261), Val(262), Val(263), Val(264), Val(265), Val(266), Val(267), Val(268), Val(269),
          Val(270), Val(271), Val(272), Val(273), Val(274), Val(275), Val(276), Val(277), Val(278),
          Val(279), Val(280), Val(281), Val(282), Val(283), Val(284), Val(285), Val(286), Val(287),
          Val(288), Val(289), Val(290), Val(291), Val(292), Val(293), Val(294), Val(295), Val(296),
          Val(297), Val(298), Val(299), Val(300), Val(301), Val(302), Val(303), Val(304), Val(305),
          Val(306), Val(307), Val(308), Val(309), Val(310), Val(311), Val(312), Val(313), Val(314),
          Val(315), Val(316), Val(317), Val(318), Val(319), Val(320), Val(321), Val(322), Val(323),
          Val(324), Val(325), Val(326), Val(327), Val(328), Val(329), Val(330), Val(331), Val(332),
          Val(333), Val(334), Val(335), Val(336), Val(337), Val(338), Val(339), Val(340), Val(341),
          Val(342), Val(343), Val(344), Val(345), Val(346), Val(347), Val(348), Val(349), Val(350),
          Val(351), Val(352), Val(353), Val(354), Val(355), Val(356), Val(357), Val(358), Val(359),
          Val(360), Val(361), Val(362), Val(363), Val(364), Val(365), Val(366), Val(367), Val(368),
          Val(369), Val(370), Val(371), Val(372), Val(373), Val(374), Val(375), Val(376), Val(377),
          Val(378), Val(379), Val(380), Val(381), Val(382), Val(383), Val(384), Val(385), Val(386),
          Val(387), Val(388), Val(389), Val(390), Val(391), Val(392), Val(393), Val(394), Val(395),
          Val(396), Val(397), Val(398), Val(399), Val(400), Val(401), Val(402), Val(403), Val(404),
          Val(405), Val(406), Val(407), Val(408), Val(409), Val(410), Val(411), Val(412), Val(413),
          Val(414), Val(415), Val(416), Val(417), Val(418), Val(419), Val(420), Val(421), Val(422),
          Val(423), Val(424), Val(425), Val(426), Val(427), Val(428), Val(429), Val(430), Val(431),
          Val(432), Val(433), Val(434), Val(435), Val(436), Val(437), Val(438), Val(439), Val(440),
          Val(441), Val(442), Val(443), Val(444), Val(445), Val(446), Val(447), Val(448), Val(449),
          Val(450), Val(451), Val(452), Val(453), Val(454), Val(455), Val(456), Val(457), Val(458),
          Val(459), Val(460), Val(461), Val(462), Val(463), Val(464), Val(465), Val(466), Val(467),
          Val(468), Val(469), Val(470), Val(471), Val(472), Val(473), Val(474), Val(475), Val(476),
          Val(477), Val(478), Val(479), Val(480), Val(481), Val(482), Val(483), Val(484), Val(485),
          Val(486), Val(487), Val(488), Val(489), Val(490), Val(491), Val(492), Val(493), Val(494),
          Val(495), Val(496), Val(497), Val(498), Val(499), Val(500), Val(501), Val(502), Val(503),
          Val(504), Val(505), Val(506), Val(507), Val(508), Val(509), Val(510), Val(511), Val(512),
          Val(513), Val(514), Val(515), Val(516), Val(517), Val(518), Val(519), Val(520), Val(521),
          Val(522), Val(523), Val(524), Val(525), Val(526), Val(527), Val(528), Val(529), Val(530),
          Val(531), Val(532), Val(533), Val(534), Val(535), Val(536), Val(537), Val(538), Val(539),
          Val(540), Val(541), Val(542), Val(543), Val(544), Val(545), Val(546), Val(547), Val(548),
          Val(549), Val(550), Val(551), Val(552), Val(553), Val(554), Val(555), Val(556), Val(557),
          Val(558), Val(559), Val(560), Val(561), Val(562), Val(563), Val(564), Val(565), Val(566),
          Val(567), Val(568), Val(569), Val(570), Val(571), Val(572), Val(573), Val(574), Val(575),
          Val(576), Val(577), Val(578), Val(579), Val(580), Val(581), Val(582), Val(583), Val(584),
          Val(585), Val(586), Val(587), Val(588), Val(589), Val(590), Val(591), Val(592), Val(593),
          Val(594), Val(595), Val(596), Val(597), Val(598), Val(599), Val(600), Val(601), Val(602),
          Val(603), Val(604), Val(605), Val(606), Val(607), Val(608), Val(609), Val(610), Val(611),
          Val(612), Val(613), Val(614), Val(615), Val(616), Val(617), Val(618), Val(619), Val(620),
          Val(621), Val(622), Val(623), Val(624), Val(625), Val(626), Val(627), Val(628), Val(629),
          Val(630), Val(631), Val(632), Val(633), Val(634), Val(635), Val(636), Val(637), Val(638),
          Val(639), Val(640), Val(641), Val(642), Val(643), Val(644), Val(645), Val(646), Val(647),
          Val(648), Val(649), Val(650), Val(651), Val(652), Val(653), Val(654), Val(655), Val(656),
          Val(657), Val(658), Val(659), Val(660), Val(661), Val(662), Val(663), Val(664), Val(665),
          Val(666), Val(667), Val(668), Val(669), Val(670), Val(671), Val(672), Val(673), Val(674),
          Val(675), Val(676), Val(677), Val(678), Val(679), Val(680), Val(681), Val(682), Val(683),
          Val(684), Val(685), Val(686), Val(687), Val(688), Val(689), Val(690), Val(691), Val(692),
          Val(693), Val(694), Val(695), Val(696), Val(697), Val(698), Val(699), Val(700), Val(701),
          Val(702), Val(703), Val(704), Val(705), Val(706), Val(707), Val(708), Val(709), Val(710),
          Val(711), Val(712), Val(713), Val(714), Val(715), Val(716), Val(717), Val(718), Val(719),
          Val(720), Val(721), Val(722), Val(723), Val(724), Val(725), Val(726), Val(727), Val(728),
          Val(729), Val(730), Val(731), Val(732), Val(733), Val(734), Val(735), Val(736), Val(737),
          Val(738), Val(739), Val(740), Val(741), Val(742), Val(743), Val(744), Val(745), Val(746),
          Val(747), Val(748), Val(749), Val(750), Val(751), Val(752), Val(753), Val(754), Val(755),
          Val(756), Val(757), Val(758), Val(759), Val(760), Val(761), Val(762), Val(763), Val(764),
          Val(765), Val(766), Val(767), Val(768), Val(769), Val(770), Val(771), Val(772), Val(773),
          Val(774), Val(775), Val(776), Val(777), Val(778), Val(779), Val(780), Val(781), Val(782),
          Val(783), Val(784), Val(785), Val(786), Val(787), Val(788), Val(789), Val(790), Val(791),
          Val(792), Val(793), Val(794), Val(795), Val(796), Val(797), Val(798), Val(799)},
      LAYOUT_LOOKUP(layout1, _super),
      ([&](Val800Array::value_type x803,
           BoundLayout<NondetRegLayout800LayoutArray::value_type> x804) {
        NondetRegStruct x805 =
            exec_NondetBitReg(ctx,
                              DivStruct800Array{x2,
                                                DivStruct{._super = x3},
                                                DivStruct{._super = x4},
                                                DivStruct{._super = x5},
                                                DivStruct{._super = x6},
                                                DivStruct{._super = x7},
                                                DivStruct{._super = x8},
                                                DivStruct{._super = x9},
                                                DivStruct{._super = x10},
                                                DivStruct{._super = x11},
                                                DivStruct{._super = x12},
                                                DivStruct{._super = x13},
                                                DivStruct{._super = x14},
                                                DivStruct{._super = x15},
                                                DivStruct{._super = x16},
                                                DivStruct{._super = x17},
                                                x18,
                                                DivStruct{._super = x19},
                                                DivStruct{._super = x20},
                                                DivStruct{._super = x21},
                                                DivStruct{._super = x22},
                                                DivStruct{._super = x23},
                                                DivStruct{._super = x24},
                                                DivStruct{._super = x25},
                                                DivStruct{._super = x26},
                                                DivStruct{._super = x27},
                                                DivStruct{._super = x28},
                                                DivStruct{._super = x29},
                                                DivStruct{._super = x30},
                                                DivStruct{._super = x31},
                                                DivStruct{._super = x32},
                                                DivStruct{._super = x33},
                                                x34,
                                                DivStruct{._super = x35},
                                                DivStruct{._super = x36},
                                                DivStruct{._super = x37},
                                                DivStruct{._super = x38},
                                                DivStruct{._super = x39},
                                                DivStruct{._super = x40},
                                                DivStruct{._super = x41},
                                                DivStruct{._super = x42},
                                                DivStruct{._super = x43},
                                                DivStruct{._super = x44},
                                                DivStruct{._super = x45},
                                                DivStruct{._super = x46},
                                                DivStruct{._super = x47},
                                                DivStruct{._super = x48},
                                                DivStruct{._super = x49},
                                                x50,
                                                DivStruct{._super = x51},
                                                DivStruct{._super = x52},
                                                DivStruct{._super = x53},
                                                DivStruct{._super = x54},
                                                DivStruct{._super = x55},
                                                DivStruct{._super = x56},
                                                DivStruct{._super = x57},
                                                DivStruct{._super = x58},
                                                DivStruct{._super = x59},
                                                DivStruct{._super = x60},
                                                DivStruct{._super = x61},
                                                DivStruct{._super = x62},
                                                DivStruct{._super = x63},
                                                DivStruct{._super = x64},
                                                DivStruct{._super = x65},
                                                x66,
                                                DivStruct{._super = x67},
                                                DivStruct{._super = x68},
                                                DivStruct{._super = x69},
                                                DivStruct{._super = x70},
                                                DivStruct{._super = x71},
                                                DivStruct{._super = x72},
                                                DivStruct{._super = x73},
                                                DivStruct{._super = x74},
                                                DivStruct{._super = x75},
                                                DivStruct{._super = x76},
                                                DivStruct{._super = x77},
                                                DivStruct{._super = x78},
                                                DivStruct{._super = x79},
                                                DivStruct{._super = x80},
                                                DivStruct{._super = x81},
                                                x82,
                                                DivStruct{._super = x83},
                                                DivStruct{._super = x84},
                                                DivStruct{._super = x85},
                                                DivStruct{._super = x86},
                                                DivStruct{._super = x87},
                                                DivStruct{._super = x88},
                                                DivStruct{._super = x89},
                                                DivStruct{._super = x90},
                                                DivStruct{._super = x91},
                                                DivStruct{._super = x92},
                                                DivStruct{._super = x93},
                                                DivStruct{._super = x94},
                                                DivStruct{._super = x95},
                                                DivStruct{._super = x96},
                                                DivStruct{._super = x97},
                                                x98,
                                                DivStruct{._super = x99},
                                                DivStruct{._super = x100},
                                                DivStruct{._super = x101},
                                                DivStruct{._super = x102},
                                                DivStruct{._super = x103},
                                                DivStruct{._super = x104},
                                                DivStruct{._super = x105},
                                                DivStruct{._super = x106},
                                                DivStruct{._super = x107},
                                                DivStruct{._super = x108},
                                                DivStruct{._super = x109},
                                                DivStruct{._super = x110},
                                                DivStruct{._super = x111},
                                                DivStruct{._super = x112},
                                                DivStruct{._super = x113},
                                                x114,
                                                DivStruct{._super = x115},
                                                DivStruct{._super = x116},
                                                DivStruct{._super = x117},
                                                DivStruct{._super = x118},
                                                DivStruct{._super = x119},
                                                DivStruct{._super = x120},
                                                DivStruct{._super = x121},
                                                DivStruct{._super = x122},
                                                DivStruct{._super = x123},
                                                DivStruct{._super = x124},
                                                DivStruct{._super = x125},
                                                DivStruct{._super = x126},
                                                DivStruct{._super = x127},
                                                DivStruct{._super = x128},
                                                DivStruct{._super = x129},
                                                x130,
                                                DivStruct{._super = x131},
                                                DivStruct{._super = x132},
                                                DivStruct{._super = x133},
                                                DivStruct{._super = x134},
                                                DivStruct{._super = x135},
                                                DivStruct{._super = x136},
                                                DivStruct{._super = x137},
                                                DivStruct{._super = x138},
                                                DivStruct{._super = x139},
                                                DivStruct{._super = x140},
                                                DivStruct{._super = x141},
                                                DivStruct{._super = x142},
                                                DivStruct{._super = x143},
                                                DivStruct{._super = x144},
                                                DivStruct{._super = x145},
                                                x146,
                                                DivStruct{._super = x147},
                                                DivStruct{._super = x148},
                                                DivStruct{._super = x149},
                                                DivStruct{._super = x150},
                                                DivStruct{._super = x151},
                                                DivStruct{._super = x152},
                                                DivStruct{._super = x153},
                                                DivStruct{._super = x154},
                                                DivStruct{._super = x155},
                                                DivStruct{._super = x156},
                                                DivStruct{._super = x157},
                                                DivStruct{._super = x158},
                                                DivStruct{._super = x159},
                                                DivStruct{._super = x160},
                                                DivStruct{._super = x161},
                                                x162,
                                                DivStruct{._super = x163},
                                                DivStruct{._super = x164},
                                                DivStruct{._super = x165},
                                                DivStruct{._super = x166},
                                                DivStruct{._super = x167},
                                                DivStruct{._super = x168},
                                                DivStruct{._super = x169},
                                                DivStruct{._super = x170},
                                                DivStruct{._super = x171},
                                                DivStruct{._super = x172},
                                                DivStruct{._super = x173},
                                                DivStruct{._super = x174},
                                                DivStruct{._super = x175},
                                                DivStruct{._super = x176},
                                                DivStruct{._super = x177},
                                                x178,
                                                DivStruct{._super = x179},
                                                DivStruct{._super = x180},
                                                DivStruct{._super = x181},
                                                DivStruct{._super = x182},
                                                DivStruct{._super = x183},
                                                DivStruct{._super = x184},
                                                DivStruct{._super = x185},
                                                DivStruct{._super = x186},
                                                DivStruct{._super = x187},
                                                DivStruct{._super = x188},
                                                DivStruct{._super = x189},
                                                DivStruct{._super = x190},
                                                DivStruct{._super = x191},
                                                DivStruct{._super = x192},
                                                DivStruct{._super = x193},
                                                x194,
                                                DivStruct{._super = x195},
                                                DivStruct{._super = x196},
                                                DivStruct{._super = x197},
                                                DivStruct{._super = x198},
                                                DivStruct{._super = x199},
                                                DivStruct{._super = x200},
                                                DivStruct{._super = x201},
                                                DivStruct{._super = x202},
                                                DivStruct{._super = x203},
                                                DivStruct{._super = x204},
                                                DivStruct{._super = x205},
                                                DivStruct{._super = x206},
                                                DivStruct{._super = x207},
                                                DivStruct{._super = x208},
                                                DivStruct{._super = x209},
                                                x210,
                                                DivStruct{._super = x211},
                                                DivStruct{._super = x212},
                                                DivStruct{._super = x213},
                                                DivStruct{._super = x214},
                                                DivStruct{._super = x215},
                                                DivStruct{._super = x216},
                                                DivStruct{._super = x217},
                                                DivStruct{._super = x218},
                                                DivStruct{._super = x219},
                                                DivStruct{._super = x220},
                                                DivStruct{._super = x221},
                                                DivStruct{._super = x222},
                                                DivStruct{._super = x223},
                                                DivStruct{._super = x224},
                                                DivStruct{._super = x225},
                                                x226,
                                                DivStruct{._super = x227},
                                                DivStruct{._super = x228},
                                                DivStruct{._super = x229},
                                                DivStruct{._super = x230},
                                                DivStruct{._super = x231},
                                                DivStruct{._super = x232},
                                                DivStruct{._super = x233},
                                                DivStruct{._super = x234},
                                                DivStruct{._super = x235},
                                                DivStruct{._super = x236},
                                                DivStruct{._super = x237},
                                                DivStruct{._super = x238},
                                                DivStruct{._super = x239},
                                                DivStruct{._super = x240},
                                                DivStruct{._super = x241},
                                                x242,
                                                DivStruct{._super = x243},
                                                DivStruct{._super = x244},
                                                DivStruct{._super = x245},
                                                DivStruct{._super = x246},
                                                DivStruct{._super = x247},
                                                DivStruct{._super = x248},
                                                DivStruct{._super = x249},
                                                DivStruct{._super = x250},
                                                DivStruct{._super = x251},
                                                DivStruct{._super = x252},
                                                DivStruct{._super = x253},
                                                DivStruct{._super = x254},
                                                DivStruct{._super = x255},
                                                DivStruct{._super = x256},
                                                DivStruct{._super = x257},
                                                x258,
                                                DivStruct{._super = x259},
                                                DivStruct{._super = x260},
                                                DivStruct{._super = x261},
                                                DivStruct{._super = x262},
                                                DivStruct{._super = x263},
                                                DivStruct{._super = x264},
                                                DivStruct{._super = x265},
                                                DivStruct{._super = x266},
                                                DivStruct{._super = x267},
                                                DivStruct{._super = x268},
                                                DivStruct{._super = x269},
                                                DivStruct{._super = x270},
                                                DivStruct{._super = x271},
                                                DivStruct{._super = x272},
                                                DivStruct{._super = x273},
                                                x274,
                                                DivStruct{._super = x275},
                                                DivStruct{._super = x276},
                                                DivStruct{._super = x277},
                                                DivStruct{._super = x278},
                                                DivStruct{._super = x279},
                                                DivStruct{._super = x280},
                                                DivStruct{._super = x281},
                                                DivStruct{._super = x282},
                                                DivStruct{._super = x283},
                                                DivStruct{._super = x284},
                                                DivStruct{._super = x285},
                                                DivStruct{._super = x286},
                                                DivStruct{._super = x287},
                                                DivStruct{._super = x288},
                                                DivStruct{._super = x289},
                                                x290,
                                                DivStruct{._super = x291},
                                                DivStruct{._super = x292},
                                                DivStruct{._super = x293},
                                                DivStruct{._super = x294},
                                                DivStruct{._super = x295},
                                                DivStruct{._super = x296},
                                                DivStruct{._super = x297},
                                                DivStruct{._super = x298},
                                                DivStruct{._super = x299},
                                                DivStruct{._super = x300},
                                                DivStruct{._super = x301},
                                                DivStruct{._super = x302},
                                                DivStruct{._super = x303},
                                                DivStruct{._super = x304},
                                                DivStruct{._super = x305},
                                                x306,
                                                DivStruct{._super = x307},
                                                DivStruct{._super = x308},
                                                DivStruct{._super = x309},
                                                DivStruct{._super = x310},
                                                DivStruct{._super = x311},
                                                DivStruct{._super = x312},
                                                DivStruct{._super = x313},
                                                DivStruct{._super = x314},
                                                DivStruct{._super = x315},
                                                DivStruct{._super = x316},
                                                DivStruct{._super = x317},
                                                DivStruct{._super = x318},
                                                DivStruct{._super = x319},
                                                DivStruct{._super = x320},
                                                DivStruct{._super = x321},
                                                x322,
                                                DivStruct{._super = x323},
                                                DivStruct{._super = x324},
                                                DivStruct{._super = x325},
                                                DivStruct{._super = x326},
                                                DivStruct{._super = x327},
                                                DivStruct{._super = x328},
                                                DivStruct{._super = x329},
                                                DivStruct{._super = x330},
                                                DivStruct{._super = x331},
                                                DivStruct{._super = x332},
                                                DivStruct{._super = x333},
                                                DivStruct{._super = x334},
                                                DivStruct{._super = x335},
                                                DivStruct{._super = x336},
                                                DivStruct{._super = x337},
                                                x338,
                                                DivStruct{._super = x339},
                                                DivStruct{._super = x340},
                                                DivStruct{._super = x341},
                                                DivStruct{._super = x342},
                                                DivStruct{._super = x343},
                                                DivStruct{._super = x344},
                                                DivStruct{._super = x345},
                                                DivStruct{._super = x346},
                                                DivStruct{._super = x347},
                                                DivStruct{._super = x348},
                                                DivStruct{._super = x349},
                                                DivStruct{._super = x350},
                                                DivStruct{._super = x351},
                                                DivStruct{._super = x352},
                                                DivStruct{._super = x353},
                                                x354,
                                                DivStruct{._super = x355},
                                                DivStruct{._super = x356},
                                                DivStruct{._super = x357},
                                                DivStruct{._super = x358},
                                                DivStruct{._super = x359},
                                                DivStruct{._super = x360},
                                                DivStruct{._super = x361},
                                                DivStruct{._super = x362},
                                                DivStruct{._super = x363},
                                                DivStruct{._super = x364},
                                                DivStruct{._super = x365},
                                                DivStruct{._super = x366},
                                                DivStruct{._super = x367},
                                                DivStruct{._super = x368},
                                                DivStruct{._super = x369},
                                                x370,
                                                DivStruct{._super = x371},
                                                DivStruct{._super = x372},
                                                DivStruct{._super = x373},
                                                DivStruct{._super = x374},
                                                DivStruct{._super = x375},
                                                DivStruct{._super = x376},
                                                DivStruct{._super = x377},
                                                DivStruct{._super = x378},
                                                DivStruct{._super = x379},
                                                DivStruct{._super = x380},
                                                DivStruct{._super = x381},
                                                DivStruct{._super = x382},
                                                DivStruct{._super = x383},
                                                DivStruct{._super = x384},
                                                DivStruct{._super = x385},
                                                x386,
                                                DivStruct{._super = x387},
                                                DivStruct{._super = x388},
                                                DivStruct{._super = x389},
                                                DivStruct{._super = x390},
                                                DivStruct{._super = x391},
                                                DivStruct{._super = x392},
                                                DivStruct{._super = x393},
                                                DivStruct{._super = x394},
                                                DivStruct{._super = x395},
                                                DivStruct{._super = x396},
                                                DivStruct{._super = x397},
                                                DivStruct{._super = x398},
                                                DivStruct{._super = x399},
                                                DivStruct{._super = x400},
                                                DivStruct{._super = x401},
                                                x402,
                                                DivStruct{._super = x403},
                                                DivStruct{._super = x404},
                                                DivStruct{._super = x405},
                                                DivStruct{._super = x406},
                                                DivStruct{._super = x407},
                                                DivStruct{._super = x408},
                                                DivStruct{._super = x409},
                                                DivStruct{._super = x410},
                                                DivStruct{._super = x411},
                                                DivStruct{._super = x412},
                                                DivStruct{._super = x413},
                                                DivStruct{._super = x414},
                                                DivStruct{._super = x415},
                                                DivStruct{._super = x416},
                                                DivStruct{._super = x417},
                                                x418,
                                                DivStruct{._super = x419},
                                                DivStruct{._super = x420},
                                                DivStruct{._super = x421},
                                                DivStruct{._super = x422},
                                                DivStruct{._super = x423},
                                                DivStruct{._super = x424},
                                                DivStruct{._super = x425},
                                                DivStruct{._super = x426},
                                                DivStruct{._super = x427},
                                                DivStruct{._super = x428},
                                                DivStruct{._super = x429},
                                                DivStruct{._super = x430},
                                                DivStruct{._super = x431},
                                                DivStruct{._super = x432},
                                                DivStruct{._super = x433},
                                                x434,
                                                DivStruct{._super = x435},
                                                DivStruct{._super = x436},
                                                DivStruct{._super = x437},
                                                DivStruct{._super = x438},
                                                DivStruct{._super = x439},
                                                DivStruct{._super = x440},
                                                DivStruct{._super = x441},
                                                DivStruct{._super = x442},
                                                DivStruct{._super = x443},
                                                DivStruct{._super = x444},
                                                DivStruct{._super = x445},
                                                DivStruct{._super = x446},
                                                DivStruct{._super = x447},
                                                DivStruct{._super = x448},
                                                DivStruct{._super = x449},
                                                x450,
                                                DivStruct{._super = x451},
                                                DivStruct{._super = x452},
                                                DivStruct{._super = x453},
                                                DivStruct{._super = x454},
                                                DivStruct{._super = x455},
                                                DivStruct{._super = x456},
                                                DivStruct{._super = x457},
                                                DivStruct{._super = x458},
                                                DivStruct{._super = x459},
                                                DivStruct{._super = x460},
                                                DivStruct{._super = x461},
                                                DivStruct{._super = x462},
                                                DivStruct{._super = x463},
                                                DivStruct{._super = x464},
                                                DivStruct{._super = x465},
                                                x466,
                                                DivStruct{._super = x467},
                                                DivStruct{._super = x468},
                                                DivStruct{._super = x469},
                                                DivStruct{._super = x470},
                                                DivStruct{._super = x471},
                                                DivStruct{._super = x472},
                                                DivStruct{._super = x473},
                                                DivStruct{._super = x474},
                                                DivStruct{._super = x475},
                                                DivStruct{._super = x476},
                                                DivStruct{._super = x477},
                                                DivStruct{._super = x478},
                                                DivStruct{._super = x479},
                                                DivStruct{._super = x480},
                                                DivStruct{._super = x481},
                                                x482,
                                                DivStruct{._super = x483},
                                                DivStruct{._super = x484},
                                                DivStruct{._super = x485},
                                                DivStruct{._super = x486},
                                                DivStruct{._super = x487},
                                                DivStruct{._super = x488},
                                                DivStruct{._super = x489},
                                                DivStruct{._super = x490},
                                                DivStruct{._super = x491},
                                                DivStruct{._super = x492},
                                                DivStruct{._super = x493},
                                                DivStruct{._super = x494},
                                                DivStruct{._super = x495},
                                                DivStruct{._super = x496},
                                                DivStruct{._super = x497},
                                                x498,
                                                DivStruct{._super = x499},
                                                DivStruct{._super = x500},
                                                DivStruct{._super = x501},
                                                DivStruct{._super = x502},
                                                DivStruct{._super = x503},
                                                DivStruct{._super = x504},
                                                DivStruct{._super = x505},
                                                DivStruct{._super = x506},
                                                DivStruct{._super = x507},
                                                DivStruct{._super = x508},
                                                DivStruct{._super = x509},
                                                DivStruct{._super = x510},
                                                DivStruct{._super = x511},
                                                DivStruct{._super = x512},
                                                DivStruct{._super = x513},
                                                x514,
                                                DivStruct{._super = x515},
                                                DivStruct{._super = x516},
                                                DivStruct{._super = x517},
                                                DivStruct{._super = x518},
                                                DivStruct{._super = x519},
                                                DivStruct{._super = x520},
                                                DivStruct{._super = x521},
                                                DivStruct{._super = x522},
                                                DivStruct{._super = x523},
                                                DivStruct{._super = x524},
                                                DivStruct{._super = x525},
                                                DivStruct{._super = x526},
                                                DivStruct{._super = x527},
                                                DivStruct{._super = x528},
                                                DivStruct{._super = x529},
                                                x530,
                                                DivStruct{._super = x531},
                                                DivStruct{._super = x532},
                                                DivStruct{._super = x533},
                                                DivStruct{._super = x534},
                                                DivStruct{._super = x535},
                                                DivStruct{._super = x536},
                                                DivStruct{._super = x537},
                                                DivStruct{._super = x538},
                                                DivStruct{._super = x539},
                                                DivStruct{._super = x540},
                                                DivStruct{._super = x541},
                                                DivStruct{._super = x542},
                                                DivStruct{._super = x543},
                                                DivStruct{._super = x544},
                                                DivStruct{._super = x545},
                                                x546,
                                                DivStruct{._super = x547},
                                                DivStruct{._super = x548},
                                                DivStruct{._super = x549},
                                                DivStruct{._super = x550},
                                                DivStruct{._super = x551},
                                                DivStruct{._super = x552},
                                                DivStruct{._super = x553},
                                                DivStruct{._super = x554},
                                                DivStruct{._super = x555},
                                                DivStruct{._super = x556},
                                                DivStruct{._super = x557},
                                                DivStruct{._super = x558},
                                                DivStruct{._super = x559},
                                                DivStruct{._super = x560},
                                                DivStruct{._super = x561},
                                                x562,
                                                DivStruct{._super = x563},
                                                DivStruct{._super = x564},
                                                DivStruct{._super = x565},
                                                DivStruct{._super = x566},
                                                DivStruct{._super = x567},
                                                DivStruct{._super = x568},
                                                DivStruct{._super = x569},
                                                DivStruct{._super = x570},
                                                DivStruct{._super = x571},
                                                DivStruct{._super = x572},
                                                DivStruct{._super = x573},
                                                DivStruct{._super = x574},
                                                DivStruct{._super = x575},
                                                DivStruct{._super = x576},
                                                DivStruct{._super = x577},
                                                x578,
                                                DivStruct{._super = x579},
                                                DivStruct{._super = x580},
                                                DivStruct{._super = x581},
                                                DivStruct{._super = x582},
                                                DivStruct{._super = x583},
                                                DivStruct{._super = x584},
                                                DivStruct{._super = x585},
                                                DivStruct{._super = x586},
                                                DivStruct{._super = x587},
                                                DivStruct{._super = x588},
                                                DivStruct{._super = x589},
                                                DivStruct{._super = x590},
                                                DivStruct{._super = x591},
                                                DivStruct{._super = x592},
                                                DivStruct{._super = x593},
                                                x594,
                                                DivStruct{._super = x595},
                                                DivStruct{._super = x596},
                                                DivStruct{._super = x597},
                                                DivStruct{._super = x598},
                                                DivStruct{._super = x599},
                                                DivStruct{._super = x600},
                                                DivStruct{._super = x601},
                                                DivStruct{._super = x602},
                                                DivStruct{._super = x603},
                                                DivStruct{._super = x604},
                                                DivStruct{._super = x605},
                                                DivStruct{._super = x606},
                                                DivStruct{._super = x607},
                                                DivStruct{._super = x608},
                                                DivStruct{._super = x609},
                                                x610,
                                                DivStruct{._super = x611},
                                                DivStruct{._super = x612},
                                                DivStruct{._super = x613},
                                                DivStruct{._super = x614},
                                                DivStruct{._super = x615},
                                                DivStruct{._super = x616},
                                                DivStruct{._super = x617},
                                                DivStruct{._super = x618},
                                                DivStruct{._super = x619},
                                                DivStruct{._super = x620},
                                                DivStruct{._super = x621},
                                                DivStruct{._super = x622},
                                                DivStruct{._super = x623},
                                                DivStruct{._super = x624},
                                                DivStruct{._super = x625},
                                                x626,
                                                DivStruct{._super = x627},
                                                DivStruct{._super = x628},
                                                DivStruct{._super = x629},
                                                DivStruct{._super = x630},
                                                DivStruct{._super = x631},
                                                DivStruct{._super = x632},
                                                DivStruct{._super = x633},
                                                DivStruct{._super = x634},
                                                DivStruct{._super = x635},
                                                DivStruct{._super = x636},
                                                DivStruct{._super = x637},
                                                DivStruct{._super = x638},
                                                DivStruct{._super = x639},
                                                DivStruct{._super = x640},
                                                DivStruct{._super = x641},
                                                x642,
                                                DivStruct{._super = x643},
                                                DivStruct{._super = x644},
                                                DivStruct{._super = x645},
                                                DivStruct{._super = x646},
                                                DivStruct{._super = x647},
                                                DivStruct{._super = x648},
                                                DivStruct{._super = x649},
                                                DivStruct{._super = x650},
                                                DivStruct{._super = x651},
                                                DivStruct{._super = x652},
                                                DivStruct{._super = x653},
                                                DivStruct{._super = x654},
                                                DivStruct{._super = x655},
                                                DivStruct{._super = x656},
                                                DivStruct{._super = x657},
                                                x658,
                                                DivStruct{._super = x659},
                                                DivStruct{._super = x660},
                                                DivStruct{._super = x661},
                                                DivStruct{._super = x662},
                                                DivStruct{._super = x663},
                                                DivStruct{._super = x664},
                                                DivStruct{._super = x665},
                                                DivStruct{._super = x666},
                                                DivStruct{._super = x667},
                                                DivStruct{._super = x668},
                                                DivStruct{._super = x669},
                                                DivStruct{._super = x670},
                                                DivStruct{._super = x671},
                                                DivStruct{._super = x672},
                                                DivStruct{._super = x673},
                                                x674,
                                                DivStruct{._super = x675},
                                                DivStruct{._super = x676},
                                                DivStruct{._super = x677},
                                                DivStruct{._super = x678},
                                                DivStruct{._super = x679},
                                                DivStruct{._super = x680},
                                                DivStruct{._super = x681},
                                                DivStruct{._super = x682},
                                                DivStruct{._super = x683},
                                                DivStruct{._super = x684},
                                                DivStruct{._super = x685},
                                                DivStruct{._super = x686},
                                                DivStruct{._super = x687},
                                                DivStruct{._super = x688},
                                                DivStruct{._super = x689},
                                                x690,
                                                DivStruct{._super = x691},
                                                DivStruct{._super = x692},
                                                DivStruct{._super = x693},
                                                DivStruct{._super = x694},
                                                DivStruct{._super = x695},
                                                DivStruct{._super = x696},
                                                DivStruct{._super = x697},
                                                DivStruct{._super = x698},
                                                DivStruct{._super = x699},
                                                DivStruct{._super = x700},
                                                DivStruct{._super = x701},
                                                DivStruct{._super = x702},
                                                DivStruct{._super = x703},
                                                DivStruct{._super = x704},
                                                DivStruct{._super = x705},
                                                x706,
                                                DivStruct{._super = x707},
                                                DivStruct{._super = x708},
                                                DivStruct{._super = x709},
                                                DivStruct{._super = x710},
                                                DivStruct{._super = x711},
                                                DivStruct{._super = x712},
                                                DivStruct{._super = x713},
                                                DivStruct{._super = x714},
                                                DivStruct{._super = x715},
                                                DivStruct{._super = x716},
                                                DivStruct{._super = x717},
                                                DivStruct{._super = x718},
                                                DivStruct{._super = x719},
                                                DivStruct{._super = x720},
                                                DivStruct{._super = x721},
                                                x722,
                                                DivStruct{._super = x723},
                                                DivStruct{._super = x724},
                                                DivStruct{._super = x725},
                                                DivStruct{._super = x726},
                                                DivStruct{._super = x727},
                                                DivStruct{._super = x728},
                                                DivStruct{._super = x729},
                                                DivStruct{._super = x730},
                                                DivStruct{._super = x731},
                                                DivStruct{._super = x732},
                                                DivStruct{._super = x733},
                                                DivStruct{._super = x734},
                                                DivStruct{._super = x735},
                                                DivStruct{._super = x736},
                                                DivStruct{._super = x737},
                                                x738,
                                                DivStruct{._super = x739},
                                                DivStruct{._super = x740},
                                                DivStruct{._super = x741},
                                                DivStruct{._super = x742},
                                                DivStruct{._super = x743},
                                                DivStruct{._super = x744},
                                                DivStruct{._super = x745},
                                                DivStruct{._super = x746},
                                                DivStruct{._super = x747},
                                                DivStruct{._super = x748},
                                                DivStruct{._super = x749},
                                                DivStruct{._super = x750},
                                                DivStruct{._super = x751},
                                                DivStruct{._super = x752},
                                                DivStruct{._super = x753},
                                                x754,
                                                DivStruct{._super = x755},
                                                DivStruct{._super = x756},
                                                DivStruct{._super = x757},
                                                DivStruct{._super = x758},
                                                DivStruct{._super = x759},
                                                DivStruct{._super = x760},
                                                DivStruct{._super = x761},
                                                DivStruct{._super = x762},
                                                DivStruct{._super = x763},
                                                DivStruct{._super = x764},
                                                DivStruct{._super = x765},
                                                DivStruct{._super = x766},
                                                DivStruct{._super = x767},
                                                DivStruct{._super = x768},
                                                DivStruct{._super = x769},
                                                x770,
                                                DivStruct{._super = x771},
                                                DivStruct{._super = x772},
                                                DivStruct{._super = x773},
                                                DivStruct{._super = x774},
                                                DivStruct{._super = x775},
                                                DivStruct{._super = x776},
                                                DivStruct{._super = x777},
                                                DivStruct{._super = x778},
                                                DivStruct{._super = x779},
                                                DivStruct{._super = x780},
                                                DivStruct{._super = x781},
                                                DivStruct{._super = x782},
                                                DivStruct{._super = x783},
                                                DivStruct{._super = x784},
                                                DivStruct{._super = x785},
                                                x786,
                                                DivStruct{._super = x787},
                                                DivStruct{._super = x788},
                                                DivStruct{._super = x789},
                                                DivStruct{._super = x790},
                                                DivStruct{._super = x791},
                                                DivStruct{._super = x792},
                                                DivStruct{._super = x793},
                                                DivStruct{._super = x794},
                                                DivStruct{._super = x795},
                                                DivStruct{._super = x796},
                                                DivStruct{._super = x797},
                                                DivStruct{._super = x798},
                                                DivStruct{._super = x799},
                                                DivStruct{._super = x800},
                                                DivStruct{._super = x801}}[to_size_t(x803)]
                                  ._super,
                              x804);
        return UnpackReg_800__16__SuperStruct{._super = x805};
      }));
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:49)
  Val x806 = (x802[1]._super._super * Val(2));
  Val x807 = (x802[2]._super._super * Val(4));
  Val x808 = (x802[3]._super._super * Val(8));
  Val x809 = (x802[4]._super._super * Val(16));
  Val x810 = (x802[5]._super._super * Val(32));
  Val x811 = (x802[6]._super._super * Val(64));
  Val x812 = (x802[7]._super._super * Val(128));
  Val x813 = (x802[8]._super._super * Val(256));
  Val x814 = (x802[9]._super._super * Val(512));
  Val x815 = (x802[10]._super._super * Val(1024));
  Val x816 = (x802[11]._super._super * Val(2048));
  Val x817 = (x802[12]._super._super * Val(4096));
  Val x818 = (x802[13]._super._super * Val(8192));
  Val x819 = (x802[14]._super._super * Val(16384));
  Val x820 = (x802[15]._super._super * Val(32768));
  Val x821 = (x802[0]._super._super + x806);
  Val x822 = (((x821 + x807) + x808) + x809);
  Val x823 = (((x822 + x810) + x811) + x812);
  Val x824 = (((x823 + x813) + x814) + x815);
  Val x825 = (((x824 + x816) + x817) + x818);
  Val x826 = (x802[17]._super._super * Val(2));
  Val x827 = (x802[18]._super._super * Val(4));
  Val x828 = (x802[19]._super._super * Val(8));
  Val x829 = (x802[20]._super._super * Val(16));
  Val x830 = (x802[21]._super._super * Val(32));
  Val x831 = (x802[22]._super._super * Val(64));
  Val x832 = (x802[23]._super._super * Val(128));
  Val x833 = (x802[24]._super._super * Val(256));
  Val x834 = (x802[25]._super._super * Val(512));
  Val x835 = (x802[26]._super._super * Val(1024));
  Val x836 = (x802[27]._super._super * Val(2048));
  Val x837 = (x802[28]._super._super * Val(4096));
  Val x838 = (x802[29]._super._super * Val(8192));
  Val x839 = (x802[30]._super._super * Val(16384));
  Val x840 = (x802[31]._super._super * Val(32768));
  Val x841 = (x802[16]._super._super + x826);
  Val x842 = (((x841 + x827) + x828) + x829);
  Val x843 = (((x842 + x830) + x831) + x832);
  Val x844 = (((x843 + x833) + x834) + x835);
  Val x845 = (((x844 + x836) + x837) + x838);
  Val x846 = (x802[33]._super._super * Val(2));
  Val x847 = (x802[34]._super._super * Val(4));
  Val x848 = (x802[35]._super._super * Val(8));
  Val x849 = (x802[36]._super._super * Val(16));
  Val x850 = (x802[37]._super._super * Val(32));
  Val x851 = (x802[38]._super._super * Val(64));
  Val x852 = (x802[39]._super._super * Val(128));
  Val x853 = (x802[40]._super._super * Val(256));
  Val x854 = (x802[41]._super._super * Val(512));
  Val x855 = (x802[42]._super._super * Val(1024));
  Val x856 = (x802[43]._super._super * Val(2048));
  Val x857 = (x802[44]._super._super * Val(4096));
  Val x858 = (x802[45]._super._super * Val(8192));
  Val x859 = (x802[46]._super._super * Val(16384));
  Val x860 = (x802[47]._super._super * Val(32768));
  Val x861 = (x802[32]._super._super + x846);
  Val x862 = (((x861 + x847) + x848) + x849);
  Val x863 = (((x862 + x850) + x851) + x852);
  Val x864 = (((x863 + x853) + x854) + x855);
  Val x865 = (((x864 + x856) + x857) + x858);
  Val x866 = (x802[49]._super._super * Val(2));
  Val x867 = (x802[50]._super._super * Val(4));
  Val x868 = (x802[51]._super._super * Val(8));
  Val x869 = (x802[52]._super._super * Val(16));
  Val x870 = (x802[53]._super._super * Val(32));
  Val x871 = (x802[54]._super._super * Val(64));
  Val x872 = (x802[55]._super._super * Val(128));
  Val x873 = (x802[56]._super._super * Val(256));
  Val x874 = (x802[57]._super._super * Val(512));
  Val x875 = (x802[58]._super._super * Val(1024));
  Val x876 = (x802[59]._super._super * Val(2048));
  Val x877 = (x802[60]._super._super * Val(4096));
  Val x878 = (x802[61]._super._super * Val(8192));
  Val x879 = (x802[62]._super._super * Val(16384));
  Val x880 = (x802[63]._super._super * Val(32768));
  Val x881 = (x802[48]._super._super + x866);
  Val x882 = (((x881 + x867) + x868) + x869);
  Val x883 = (((x882 + x870) + x871) + x872);
  Val x884 = (((x883 + x873) + x874) + x875);
  Val x885 = (((x884 + x876) + x877) + x878);
  Val x886 = (x802[65]._super._super * Val(2));
  Val x887 = (x802[66]._super._super * Val(4));
  Val x888 = (x802[67]._super._super * Val(8));
  Val x889 = (x802[68]._super._super * Val(16));
  Val x890 = (x802[69]._super._super * Val(32));
  Val x891 = (x802[70]._super._super * Val(64));
  Val x892 = (x802[71]._super._super * Val(128));
  Val x893 = (x802[72]._super._super * Val(256));
  Val x894 = (x802[73]._super._super * Val(512));
  Val x895 = (x802[74]._super._super * Val(1024));
  Val x896 = (x802[75]._super._super * Val(2048));
  Val x897 = (x802[76]._super._super * Val(4096));
  Val x898 = (x802[77]._super._super * Val(8192));
  Val x899 = (x802[78]._super._super * Val(16384));
  Val x900 = (x802[79]._super._super * Val(32768));
  Val x901 = (x802[64]._super._super + x886);
  Val x902 = (((x901 + x887) + x888) + x889);
  Val x903 = (((x902 + x890) + x891) + x892);
  Val x904 = (((x903 + x893) + x894) + x895);
  Val x905 = (((x904 + x896) + x897) + x898);
  Val x906 = (x802[81]._super._super * Val(2));
  Val x907 = (x802[82]._super._super * Val(4));
  Val x908 = (x802[83]._super._super * Val(8));
  Val x909 = (x802[84]._super._super * Val(16));
  Val x910 = (x802[85]._super._super * Val(32));
  Val x911 = (x802[86]._super._super * Val(64));
  Val x912 = (x802[87]._super._super * Val(128));
  Val x913 = (x802[88]._super._super * Val(256));
  Val x914 = (x802[89]._super._super * Val(512));
  Val x915 = (x802[90]._super._super * Val(1024));
  Val x916 = (x802[91]._super._super * Val(2048));
  Val x917 = (x802[92]._super._super * Val(4096));
  Val x918 = (x802[93]._super._super * Val(8192));
  Val x919 = (x802[94]._super._super * Val(16384));
  Val x920 = (x802[95]._super._super * Val(32768));
  Val x921 = (x802[80]._super._super + x906);
  Val x922 = (((x921 + x907) + x908) + x909);
  Val x923 = (((x922 + x910) + x911) + x912);
  Val x924 = (((x923 + x913) + x914) + x915);
  Val x925 = (((x924 + x916) + x917) + x918);
  Val x926 = (x802[97]._super._super * Val(2));
  Val x927 = (x802[98]._super._super * Val(4));
  Val x928 = (x802[99]._super._super * Val(8));
  Val x929 = (x802[100]._super._super * Val(16));
  Val x930 = (x802[101]._super._super * Val(32));
  Val x931 = (x802[102]._super._super * Val(64));
  Val x932 = (x802[103]._super._super * Val(128));
  Val x933 = (x802[104]._super._super * Val(256));
  Val x934 = (x802[105]._super._super * Val(512));
  Val x935 = (x802[106]._super._super * Val(1024));
  Val x936 = (x802[107]._super._super * Val(2048));
  Val x937 = (x802[108]._super._super * Val(4096));
  Val x938 = (x802[109]._super._super * Val(8192));
  Val x939 = (x802[110]._super._super * Val(16384));
  Val x940 = (x802[111]._super._super * Val(32768));
  Val x941 = (x802[96]._super._super + x926);
  Val x942 = (((x941 + x927) + x928) + x929);
  Val x943 = (((x942 + x930) + x931) + x932);
  Val x944 = (((x943 + x933) + x934) + x935);
  Val x945 = (((x944 + x936) + x937) + x938);
  Val x946 = (x802[113]._super._super * Val(2));
  Val x947 = (x802[114]._super._super * Val(4));
  Val x948 = (x802[115]._super._super * Val(8));
  Val x949 = (x802[116]._super._super * Val(16));
  Val x950 = (x802[117]._super._super * Val(32));
  Val x951 = (x802[118]._super._super * Val(64));
  Val x952 = (x802[119]._super._super * Val(128));
  Val x953 = (x802[120]._super._super * Val(256));
  Val x954 = (x802[121]._super._super * Val(512));
  Val x955 = (x802[122]._super._super * Val(1024));
  Val x956 = (x802[123]._super._super * Val(2048));
  Val x957 = (x802[124]._super._super * Val(4096));
  Val x958 = (x802[125]._super._super * Val(8192));
  Val x959 = (x802[126]._super._super * Val(16384));
  Val x960 = (x802[127]._super._super * Val(32768));
  Val x961 = (x802[112]._super._super + x946);
  Val x962 = (((x961 + x947) + x948) + x949);
  Val x963 = (((x962 + x950) + x951) + x952);
  Val x964 = (((x963 + x953) + x954) + x955);
  Val x965 = (((x964 + x956) + x957) + x958);
  Val x966 = (x802[129]._super._super * Val(2));
  Val x967 = (x802[130]._super._super * Val(4));
  Val x968 = (x802[131]._super._super * Val(8));
  Val x969 = (x802[132]._super._super * Val(16));
  Val x970 = (x802[133]._super._super * Val(32));
  Val x971 = (x802[134]._super._super * Val(64));
  Val x972 = (x802[135]._super._super * Val(128));
  Val x973 = (x802[136]._super._super * Val(256));
  Val x974 = (x802[137]._super._super * Val(512));
  Val x975 = (x802[138]._super._super * Val(1024));
  Val x976 = (x802[139]._super._super * Val(2048));
  Val x977 = (x802[140]._super._super * Val(4096));
  Val x978 = (x802[141]._super._super * Val(8192));
  Val x979 = (x802[142]._super._super * Val(16384));
  Val x980 = (x802[143]._super._super * Val(32768));
  Val x981 = (x802[128]._super._super + x966);
  Val x982 = (((x981 + x967) + x968) + x969);
  Val x983 = (((x982 + x970) + x971) + x972);
  Val x984 = (((x983 + x973) + x974) + x975);
  Val x985 = (((x984 + x976) + x977) + x978);
  Val x986 = (x802[145]._super._super * Val(2));
  Val x987 = (x802[146]._super._super * Val(4));
  Val x988 = (x802[147]._super._super * Val(8));
  Val x989 = (x802[148]._super._super * Val(16));
  Val x990 = (x802[149]._super._super * Val(32));
  Val x991 = (x802[150]._super._super * Val(64));
  Val x992 = (x802[151]._super._super * Val(128));
  Val x993 = (x802[152]._super._super * Val(256));
  Val x994 = (x802[153]._super._super * Val(512));
  Val x995 = (x802[154]._super._super * Val(1024));
  Val x996 = (x802[155]._super._super * Val(2048));
  Val x997 = (x802[156]._super._super * Val(4096));
  Val x998 = (x802[157]._super._super * Val(8192));
  Val x999 = (x802[158]._super._super * Val(16384));
  Val x1000 = (x802[159]._super._super * Val(32768));
  Val x1001 = (x802[144]._super._super + x986);
  Val x1002 = (((x1001 + x987) + x988) + x989);
  Val x1003 = (((x1002 + x990) + x991) + x992);
  Val x1004 = (((x1003 + x993) + x994) + x995);
  Val x1005 = (((x1004 + x996) + x997) + x998);
  Val x1006 = (x802[161]._super._super * Val(2));
  Val x1007 = (x802[162]._super._super * Val(4));
  Val x1008 = (x802[163]._super._super * Val(8));
  Val x1009 = (x802[164]._super._super * Val(16));
  Val x1010 = (x802[165]._super._super * Val(32));
  Val x1011 = (x802[166]._super._super * Val(64));
  Val x1012 = (x802[167]._super._super * Val(128));
  Val x1013 = (x802[168]._super._super * Val(256));
  Val x1014 = (x802[169]._super._super * Val(512));
  Val x1015 = (x802[170]._super._super * Val(1024));
  Val x1016 = (x802[171]._super._super * Val(2048));
  Val x1017 = (x802[172]._super._super * Val(4096));
  Val x1018 = (x802[173]._super._super * Val(8192));
  Val x1019 = (x802[174]._super._super * Val(16384));
  Val x1020 = (x802[175]._super._super * Val(32768));
  Val x1021 = (x802[160]._super._super + x1006);
  Val x1022 = (((x1021 + x1007) + x1008) + x1009);
  Val x1023 = (((x1022 + x1010) + x1011) + x1012);
  Val x1024 = (((x1023 + x1013) + x1014) + x1015);
  Val x1025 = (((x1024 + x1016) + x1017) + x1018);
  Val x1026 = (x802[177]._super._super * Val(2));
  Val x1027 = (x802[178]._super._super * Val(4));
  Val x1028 = (x802[179]._super._super * Val(8));
  Val x1029 = (x802[180]._super._super * Val(16));
  Val x1030 = (x802[181]._super._super * Val(32));
  Val x1031 = (x802[182]._super._super * Val(64));
  Val x1032 = (x802[183]._super._super * Val(128));
  Val x1033 = (x802[184]._super._super * Val(256));
  Val x1034 = (x802[185]._super._super * Val(512));
  Val x1035 = (x802[186]._super._super * Val(1024));
  Val x1036 = (x802[187]._super._super * Val(2048));
  Val x1037 = (x802[188]._super._super * Val(4096));
  Val x1038 = (x802[189]._super._super * Val(8192));
  Val x1039 = (x802[190]._super._super * Val(16384));
  Val x1040 = (x802[191]._super._super * Val(32768));
  Val x1041 = (x802[176]._super._super + x1026);
  Val x1042 = (((x1041 + x1027) + x1028) + x1029);
  Val x1043 = (((x1042 + x1030) + x1031) + x1032);
  Val x1044 = (((x1043 + x1033) + x1034) + x1035);
  Val x1045 = (((x1044 + x1036) + x1037) + x1038);
  Val x1046 = (x802[193]._super._super * Val(2));
  Val x1047 = (x802[194]._super._super * Val(4));
  Val x1048 = (x802[195]._super._super * Val(8));
  Val x1049 = (x802[196]._super._super * Val(16));
  Val x1050 = (x802[197]._super._super * Val(32));
  Val x1051 = (x802[198]._super._super * Val(64));
  Val x1052 = (x802[199]._super._super * Val(128));
  Val x1053 = (x802[200]._super._super * Val(256));
  Val x1054 = (x802[201]._super._super * Val(512));
  Val x1055 = (x802[202]._super._super * Val(1024));
  Val x1056 = (x802[203]._super._super * Val(2048));
  Val x1057 = (x802[204]._super._super * Val(4096));
  Val x1058 = (x802[205]._super._super * Val(8192));
  Val x1059 = (x802[206]._super._super * Val(16384));
  Val x1060 = (x802[207]._super._super * Val(32768));
  Val x1061 = (x802[192]._super._super + x1046);
  Val x1062 = (((x1061 + x1047) + x1048) + x1049);
  Val x1063 = (((x1062 + x1050) + x1051) + x1052);
  Val x1064 = (((x1063 + x1053) + x1054) + x1055);
  Val x1065 = (((x1064 + x1056) + x1057) + x1058);
  Val x1066 = (x802[209]._super._super * Val(2));
  Val x1067 = (x802[210]._super._super * Val(4));
  Val x1068 = (x802[211]._super._super * Val(8));
  Val x1069 = (x802[212]._super._super * Val(16));
  Val x1070 = (x802[213]._super._super * Val(32));
  Val x1071 = (x802[214]._super._super * Val(64));
  Val x1072 = (x802[215]._super._super * Val(128));
  Val x1073 = (x802[216]._super._super * Val(256));
  Val x1074 = (x802[217]._super._super * Val(512));
  Val x1075 = (x802[218]._super._super * Val(1024));
  Val x1076 = (x802[219]._super._super * Val(2048));
  Val x1077 = (x802[220]._super._super * Val(4096));
  Val x1078 = (x802[221]._super._super * Val(8192));
  Val x1079 = (x802[222]._super._super * Val(16384));
  Val x1080 = (x802[223]._super._super * Val(32768));
  Val x1081 = (x802[208]._super._super + x1066);
  Val x1082 = (((x1081 + x1067) + x1068) + x1069);
  Val x1083 = (((x1082 + x1070) + x1071) + x1072);
  Val x1084 = (((x1083 + x1073) + x1074) + x1075);
  Val x1085 = (((x1084 + x1076) + x1077) + x1078);
  Val x1086 = (x802[225]._super._super * Val(2));
  Val x1087 = (x802[226]._super._super * Val(4));
  Val x1088 = (x802[227]._super._super * Val(8));
  Val x1089 = (x802[228]._super._super * Val(16));
  Val x1090 = (x802[229]._super._super * Val(32));
  Val x1091 = (x802[230]._super._super * Val(64));
  Val x1092 = (x802[231]._super._super * Val(128));
  Val x1093 = (x802[232]._super._super * Val(256));
  Val x1094 = (x802[233]._super._super * Val(512));
  Val x1095 = (x802[234]._super._super * Val(1024));
  Val x1096 = (x802[235]._super._super * Val(2048));
  Val x1097 = (x802[236]._super._super * Val(4096));
  Val x1098 = (x802[237]._super._super * Val(8192));
  Val x1099 = (x802[238]._super._super * Val(16384));
  Val x1100 = (x802[239]._super._super * Val(32768));
  Val x1101 = (x802[224]._super._super + x1086);
  Val x1102 = (((x1101 + x1087) + x1088) + x1089);
  Val x1103 = (((x1102 + x1090) + x1091) + x1092);
  Val x1104 = (((x1103 + x1093) + x1094) + x1095);
  Val x1105 = (((x1104 + x1096) + x1097) + x1098);
  Val x1106 = (x802[241]._super._super * Val(2));
  Val x1107 = (x802[242]._super._super * Val(4));
  Val x1108 = (x802[243]._super._super * Val(8));
  Val x1109 = (x802[244]._super._super * Val(16));
  Val x1110 = (x802[245]._super._super * Val(32));
  Val x1111 = (x802[246]._super._super * Val(64));
  Val x1112 = (x802[247]._super._super * Val(128));
  Val x1113 = (x802[248]._super._super * Val(256));
  Val x1114 = (x802[249]._super._super * Val(512));
  Val x1115 = (x802[250]._super._super * Val(1024));
  Val x1116 = (x802[251]._super._super * Val(2048));
  Val x1117 = (x802[252]._super._super * Val(4096));
  Val x1118 = (x802[253]._super._super * Val(8192));
  Val x1119 = (x802[254]._super._super * Val(16384));
  Val x1120 = (x802[255]._super._super * Val(32768));
  Val x1121 = (x802[240]._super._super + x1106);
  Val x1122 = (((x1121 + x1107) + x1108) + x1109);
  Val x1123 = (((x1122 + x1110) + x1111) + x1112);
  Val x1124 = (((x1123 + x1113) + x1114) + x1115);
  Val x1125 = (((x1124 + x1116) + x1117) + x1118);
  Val x1126 = (x802[257]._super._super * Val(2));
  Val x1127 = (x802[258]._super._super * Val(4));
  Val x1128 = (x802[259]._super._super * Val(8));
  Val x1129 = (x802[260]._super._super * Val(16));
  Val x1130 = (x802[261]._super._super * Val(32));
  Val x1131 = (x802[262]._super._super * Val(64));
  Val x1132 = (x802[263]._super._super * Val(128));
  Val x1133 = (x802[264]._super._super * Val(256));
  Val x1134 = (x802[265]._super._super * Val(512));
  Val x1135 = (x802[266]._super._super * Val(1024));
  Val x1136 = (x802[267]._super._super * Val(2048));
  Val x1137 = (x802[268]._super._super * Val(4096));
  Val x1138 = (x802[269]._super._super * Val(8192));
  Val x1139 = (x802[270]._super._super * Val(16384));
  Val x1140 = (x802[271]._super._super * Val(32768));
  Val x1141 = (x802[256]._super._super + x1126);
  Val x1142 = (((x1141 + x1127) + x1128) + x1129);
  Val x1143 = (((x1142 + x1130) + x1131) + x1132);
  Val x1144 = (((x1143 + x1133) + x1134) + x1135);
  Val x1145 = (((x1144 + x1136) + x1137) + x1138);
  Val x1146 = (x802[273]._super._super * Val(2));
  Val x1147 = (x802[274]._super._super * Val(4));
  Val x1148 = (x802[275]._super._super * Val(8));
  Val x1149 = (x802[276]._super._super * Val(16));
  Val x1150 = (x802[277]._super._super * Val(32));
  Val x1151 = (x802[278]._super._super * Val(64));
  Val x1152 = (x802[279]._super._super * Val(128));
  Val x1153 = (x802[280]._super._super * Val(256));
  Val x1154 = (x802[281]._super._super * Val(512));
  Val x1155 = (x802[282]._super._super * Val(1024));
  Val x1156 = (x802[283]._super._super * Val(2048));
  Val x1157 = (x802[284]._super._super * Val(4096));
  Val x1158 = (x802[285]._super._super * Val(8192));
  Val x1159 = (x802[286]._super._super * Val(16384));
  Val x1160 = (x802[287]._super._super * Val(32768));
  Val x1161 = (x802[272]._super._super + x1146);
  Val x1162 = (((x1161 + x1147) + x1148) + x1149);
  Val x1163 = (((x1162 + x1150) + x1151) + x1152);
  Val x1164 = (((x1163 + x1153) + x1154) + x1155);
  Val x1165 = (((x1164 + x1156) + x1157) + x1158);
  Val x1166 = (x802[289]._super._super * Val(2));
  Val x1167 = (x802[290]._super._super * Val(4));
  Val x1168 = (x802[291]._super._super * Val(8));
  Val x1169 = (x802[292]._super._super * Val(16));
  Val x1170 = (x802[293]._super._super * Val(32));
  Val x1171 = (x802[294]._super._super * Val(64));
  Val x1172 = (x802[295]._super._super * Val(128));
  Val x1173 = (x802[296]._super._super * Val(256));
  Val x1174 = (x802[297]._super._super * Val(512));
  Val x1175 = (x802[298]._super._super * Val(1024));
  Val x1176 = (x802[299]._super._super * Val(2048));
  Val x1177 = (x802[300]._super._super * Val(4096));
  Val x1178 = (x802[301]._super._super * Val(8192));
  Val x1179 = (x802[302]._super._super * Val(16384));
  Val x1180 = (x802[303]._super._super * Val(32768));
  Val x1181 = (x802[288]._super._super + x1166);
  Val x1182 = (((x1181 + x1167) + x1168) + x1169);
  Val x1183 = (((x1182 + x1170) + x1171) + x1172);
  Val x1184 = (((x1183 + x1173) + x1174) + x1175);
  Val x1185 = (((x1184 + x1176) + x1177) + x1178);
  Val x1186 = (x802[305]._super._super * Val(2));
  Val x1187 = (x802[306]._super._super * Val(4));
  Val x1188 = (x802[307]._super._super * Val(8));
  Val x1189 = (x802[308]._super._super * Val(16));
  Val x1190 = (x802[309]._super._super * Val(32));
  Val x1191 = (x802[310]._super._super * Val(64));
  Val x1192 = (x802[311]._super._super * Val(128));
  Val x1193 = (x802[312]._super._super * Val(256));
  Val x1194 = (x802[313]._super._super * Val(512));
  Val x1195 = (x802[314]._super._super * Val(1024));
  Val x1196 = (x802[315]._super._super * Val(2048));
  Val x1197 = (x802[316]._super._super * Val(4096));
  Val x1198 = (x802[317]._super._super * Val(8192));
  Val x1199 = (x802[318]._super._super * Val(16384));
  Val x1200 = (x802[319]._super._super * Val(32768));
  Val x1201 = (x802[304]._super._super + x1186);
  Val x1202 = (((x1201 + x1187) + x1188) + x1189);
  Val x1203 = (((x1202 + x1190) + x1191) + x1192);
  Val x1204 = (((x1203 + x1193) + x1194) + x1195);
  Val x1205 = (((x1204 + x1196) + x1197) + x1198);
  Val x1206 = (x802[321]._super._super * Val(2));
  Val x1207 = (x802[322]._super._super * Val(4));
  Val x1208 = (x802[323]._super._super * Val(8));
  Val x1209 = (x802[324]._super._super * Val(16));
  Val x1210 = (x802[325]._super._super * Val(32));
  Val x1211 = (x802[326]._super._super * Val(64));
  Val x1212 = (x802[327]._super._super * Val(128));
  Val x1213 = (x802[328]._super._super * Val(256));
  Val x1214 = (x802[329]._super._super * Val(512));
  Val x1215 = (x802[330]._super._super * Val(1024));
  Val x1216 = (x802[331]._super._super * Val(2048));
  Val x1217 = (x802[332]._super._super * Val(4096));
  Val x1218 = (x802[333]._super._super * Val(8192));
  Val x1219 = (x802[334]._super._super * Val(16384));
  Val x1220 = (x802[335]._super._super * Val(32768));
  Val x1221 = (x802[320]._super._super + x1206);
  Val x1222 = (((x1221 + x1207) + x1208) + x1209);
  Val x1223 = (((x1222 + x1210) + x1211) + x1212);
  Val x1224 = (((x1223 + x1213) + x1214) + x1215);
  Val x1225 = (((x1224 + x1216) + x1217) + x1218);
  Val x1226 = (x802[337]._super._super * Val(2));
  Val x1227 = (x802[338]._super._super * Val(4));
  Val x1228 = (x802[339]._super._super * Val(8));
  Val x1229 = (x802[340]._super._super * Val(16));
  Val x1230 = (x802[341]._super._super * Val(32));
  Val x1231 = (x802[342]._super._super * Val(64));
  Val x1232 = (x802[343]._super._super * Val(128));
  Val x1233 = (x802[344]._super._super * Val(256));
  Val x1234 = (x802[345]._super._super * Val(512));
  Val x1235 = (x802[346]._super._super * Val(1024));
  Val x1236 = (x802[347]._super._super * Val(2048));
  Val x1237 = (x802[348]._super._super * Val(4096));
  Val x1238 = (x802[349]._super._super * Val(8192));
  Val x1239 = (x802[350]._super._super * Val(16384));
  Val x1240 = (x802[351]._super._super * Val(32768));
  Val x1241 = (x802[336]._super._super + x1226);
  Val x1242 = (((x1241 + x1227) + x1228) + x1229);
  Val x1243 = (((x1242 + x1230) + x1231) + x1232);
  Val x1244 = (((x1243 + x1233) + x1234) + x1235);
  Val x1245 = (((x1244 + x1236) + x1237) + x1238);
  Val x1246 = (x802[353]._super._super * Val(2));
  Val x1247 = (x802[354]._super._super * Val(4));
  Val x1248 = (x802[355]._super._super * Val(8));
  Val x1249 = (x802[356]._super._super * Val(16));
  Val x1250 = (x802[357]._super._super * Val(32));
  Val x1251 = (x802[358]._super._super * Val(64));
  Val x1252 = (x802[359]._super._super * Val(128));
  Val x1253 = (x802[360]._super._super * Val(256));
  Val x1254 = (x802[361]._super._super * Val(512));
  Val x1255 = (x802[362]._super._super * Val(1024));
  Val x1256 = (x802[363]._super._super * Val(2048));
  Val x1257 = (x802[364]._super._super * Val(4096));
  Val x1258 = (x802[365]._super._super * Val(8192));
  Val x1259 = (x802[366]._super._super * Val(16384));
  Val x1260 = (x802[367]._super._super * Val(32768));
  Val x1261 = (x802[352]._super._super + x1246);
  Val x1262 = (((x1261 + x1247) + x1248) + x1249);
  Val x1263 = (((x1262 + x1250) + x1251) + x1252);
  Val x1264 = (((x1263 + x1253) + x1254) + x1255);
  Val x1265 = (((x1264 + x1256) + x1257) + x1258);
  Val x1266 = (x802[369]._super._super * Val(2));
  Val x1267 = (x802[370]._super._super * Val(4));
  Val x1268 = (x802[371]._super._super * Val(8));
  Val x1269 = (x802[372]._super._super * Val(16));
  Val x1270 = (x802[373]._super._super * Val(32));
  Val x1271 = (x802[374]._super._super * Val(64));
  Val x1272 = (x802[375]._super._super * Val(128));
  Val x1273 = (x802[376]._super._super * Val(256));
  Val x1274 = (x802[377]._super._super * Val(512));
  Val x1275 = (x802[378]._super._super * Val(1024));
  Val x1276 = (x802[379]._super._super * Val(2048));
  Val x1277 = (x802[380]._super._super * Val(4096));
  Val x1278 = (x802[381]._super._super * Val(8192));
  Val x1279 = (x802[382]._super._super * Val(16384));
  Val x1280 = (x802[383]._super._super * Val(32768));
  Val x1281 = (x802[368]._super._super + x1266);
  Val x1282 = (((x1281 + x1267) + x1268) + x1269);
  Val x1283 = (((x1282 + x1270) + x1271) + x1272);
  Val x1284 = (((x1283 + x1273) + x1274) + x1275);
  Val x1285 = (((x1284 + x1276) + x1277) + x1278);
  Val x1286 = (x802[385]._super._super * Val(2));
  Val x1287 = (x802[386]._super._super * Val(4));
  Val x1288 = (x802[387]._super._super * Val(8));
  Val x1289 = (x802[388]._super._super * Val(16));
  Val x1290 = (x802[389]._super._super * Val(32));
  Val x1291 = (x802[390]._super._super * Val(64));
  Val x1292 = (x802[391]._super._super * Val(128));
  Val x1293 = (x802[392]._super._super * Val(256));
  Val x1294 = (x802[393]._super._super * Val(512));
  Val x1295 = (x802[394]._super._super * Val(1024));
  Val x1296 = (x802[395]._super._super * Val(2048));
  Val x1297 = (x802[396]._super._super * Val(4096));
  Val x1298 = (x802[397]._super._super * Val(8192));
  Val x1299 = (x802[398]._super._super * Val(16384));
  Val x1300 = (x802[399]._super._super * Val(32768));
  Val x1301 = (x802[384]._super._super + x1286);
  Val x1302 = (((x1301 + x1287) + x1288) + x1289);
  Val x1303 = (((x1302 + x1290) + x1291) + x1292);
  Val x1304 = (((x1303 + x1293) + x1294) + x1295);
  Val x1305 = (((x1304 + x1296) + x1297) + x1298);
  Val x1306 = (x802[401]._super._super * Val(2));
  Val x1307 = (x802[402]._super._super * Val(4));
  Val x1308 = (x802[403]._super._super * Val(8));
  Val x1309 = (x802[404]._super._super * Val(16));
  Val x1310 = (x802[405]._super._super * Val(32));
  Val x1311 = (x802[406]._super._super * Val(64));
  Val x1312 = (x802[407]._super._super * Val(128));
  Val x1313 = (x802[408]._super._super * Val(256));
  Val x1314 = (x802[409]._super._super * Val(512));
  Val x1315 = (x802[410]._super._super * Val(1024));
  Val x1316 = (x802[411]._super._super * Val(2048));
  Val x1317 = (x802[412]._super._super * Val(4096));
  Val x1318 = (x802[413]._super._super * Val(8192));
  Val x1319 = (x802[414]._super._super * Val(16384));
  Val x1320 = (x802[415]._super._super * Val(32768));
  Val x1321 = (x802[400]._super._super + x1306);
  Val x1322 = (((x1321 + x1307) + x1308) + x1309);
  Val x1323 = (((x1322 + x1310) + x1311) + x1312);
  Val x1324 = (((x1323 + x1313) + x1314) + x1315);
  Val x1325 = (((x1324 + x1316) + x1317) + x1318);
  Val x1326 = (x802[417]._super._super * Val(2));
  Val x1327 = (x802[418]._super._super * Val(4));
  Val x1328 = (x802[419]._super._super * Val(8));
  Val x1329 = (x802[420]._super._super * Val(16));
  Val x1330 = (x802[421]._super._super * Val(32));
  Val x1331 = (x802[422]._super._super * Val(64));
  Val x1332 = (x802[423]._super._super * Val(128));
  Val x1333 = (x802[424]._super._super * Val(256));
  Val x1334 = (x802[425]._super._super * Val(512));
  Val x1335 = (x802[426]._super._super * Val(1024));
  Val x1336 = (x802[427]._super._super * Val(2048));
  Val x1337 = (x802[428]._super._super * Val(4096));
  Val x1338 = (x802[429]._super._super * Val(8192));
  Val x1339 = (x802[430]._super._super * Val(16384));
  Val x1340 = (x802[431]._super._super * Val(32768));
  Val x1341 = (x802[416]._super._super + x1326);
  Val x1342 = (((x1341 + x1327) + x1328) + x1329);
  Val x1343 = (((x1342 + x1330) + x1331) + x1332);
  Val x1344 = (((x1343 + x1333) + x1334) + x1335);
  Val x1345 = (((x1344 + x1336) + x1337) + x1338);
  Val x1346 = (x802[433]._super._super * Val(2));
  Val x1347 = (x802[434]._super._super * Val(4));
  Val x1348 = (x802[435]._super._super * Val(8));
  Val x1349 = (x802[436]._super._super * Val(16));
  Val x1350 = (x802[437]._super._super * Val(32));
  Val x1351 = (x802[438]._super._super * Val(64));
  Val x1352 = (x802[439]._super._super * Val(128));
  Val x1353 = (x802[440]._super._super * Val(256));
  Val x1354 = (x802[441]._super._super * Val(512));
  Val x1355 = (x802[442]._super._super * Val(1024));
  Val x1356 = (x802[443]._super._super * Val(2048));
  Val x1357 = (x802[444]._super._super * Val(4096));
  Val x1358 = (x802[445]._super._super * Val(8192));
  Val x1359 = (x802[446]._super._super * Val(16384));
  Val x1360 = (x802[447]._super._super * Val(32768));
  Val x1361 = (x802[432]._super._super + x1346);
  Val x1362 = (((x1361 + x1347) + x1348) + x1349);
  Val x1363 = (((x1362 + x1350) + x1351) + x1352);
  Val x1364 = (((x1363 + x1353) + x1354) + x1355);
  Val x1365 = (((x1364 + x1356) + x1357) + x1358);
  Val x1366 = (x802[449]._super._super * Val(2));
  Val x1367 = (x802[450]._super._super * Val(4));
  Val x1368 = (x802[451]._super._super * Val(8));
  Val x1369 = (x802[452]._super._super * Val(16));
  Val x1370 = (x802[453]._super._super * Val(32));
  Val x1371 = (x802[454]._super._super * Val(64));
  Val x1372 = (x802[455]._super._super * Val(128));
  Val x1373 = (x802[456]._super._super * Val(256));
  Val x1374 = (x802[457]._super._super * Val(512));
  Val x1375 = (x802[458]._super._super * Val(1024));
  Val x1376 = (x802[459]._super._super * Val(2048));
  Val x1377 = (x802[460]._super._super * Val(4096));
  Val x1378 = (x802[461]._super._super * Val(8192));
  Val x1379 = (x802[462]._super._super * Val(16384));
  Val x1380 = (x802[463]._super._super * Val(32768));
  Val x1381 = (x802[448]._super._super + x1366);
  Val x1382 = (((x1381 + x1367) + x1368) + x1369);
  Val x1383 = (((x1382 + x1370) + x1371) + x1372);
  Val x1384 = (((x1383 + x1373) + x1374) + x1375);
  Val x1385 = (((x1384 + x1376) + x1377) + x1378);
  Val x1386 = (x802[465]._super._super * Val(2));
  Val x1387 = (x802[466]._super._super * Val(4));
  Val x1388 = (x802[467]._super._super * Val(8));
  Val x1389 = (x802[468]._super._super * Val(16));
  Val x1390 = (x802[469]._super._super * Val(32));
  Val x1391 = (x802[470]._super._super * Val(64));
  Val x1392 = (x802[471]._super._super * Val(128));
  Val x1393 = (x802[472]._super._super * Val(256));
  Val x1394 = (x802[473]._super._super * Val(512));
  Val x1395 = (x802[474]._super._super * Val(1024));
  Val x1396 = (x802[475]._super._super * Val(2048));
  Val x1397 = (x802[476]._super._super * Val(4096));
  Val x1398 = (x802[477]._super._super * Val(8192));
  Val x1399 = (x802[478]._super._super * Val(16384));
  Val x1400 = (x802[479]._super._super * Val(32768));
  Val x1401 = (x802[464]._super._super + x1386);
  Val x1402 = (((x1401 + x1387) + x1388) + x1389);
  Val x1403 = (((x1402 + x1390) + x1391) + x1392);
  Val x1404 = (((x1403 + x1393) + x1394) + x1395);
  Val x1405 = (((x1404 + x1396) + x1397) + x1398);
  Val x1406 = (x802[481]._super._super * Val(2));
  Val x1407 = (x802[482]._super._super * Val(4));
  Val x1408 = (x802[483]._super._super * Val(8));
  Val x1409 = (x802[484]._super._super * Val(16));
  Val x1410 = (x802[485]._super._super * Val(32));
  Val x1411 = (x802[486]._super._super * Val(64));
  Val x1412 = (x802[487]._super._super * Val(128));
  Val x1413 = (x802[488]._super._super * Val(256));
  Val x1414 = (x802[489]._super._super * Val(512));
  Val x1415 = (x802[490]._super._super * Val(1024));
  Val x1416 = (x802[491]._super._super * Val(2048));
  Val x1417 = (x802[492]._super._super * Val(4096));
  Val x1418 = (x802[493]._super._super * Val(8192));
  Val x1419 = (x802[494]._super._super * Val(16384));
  Val x1420 = (x802[495]._super._super * Val(32768));
  Val x1421 = (x802[480]._super._super + x1406);
  Val x1422 = (((x1421 + x1407) + x1408) + x1409);
  Val x1423 = (((x1422 + x1410) + x1411) + x1412);
  Val x1424 = (((x1423 + x1413) + x1414) + x1415);
  Val x1425 = (((x1424 + x1416) + x1417) + x1418);
  Val x1426 = (x802[497]._super._super * Val(2));
  Val x1427 = (x802[498]._super._super * Val(4));
  Val x1428 = (x802[499]._super._super * Val(8));
  Val x1429 = (x802[500]._super._super * Val(16));
  Val x1430 = (x802[501]._super._super * Val(32));
  Val x1431 = (x802[502]._super._super * Val(64));
  Val x1432 = (x802[503]._super._super * Val(128));
  Val x1433 = (x802[504]._super._super * Val(256));
  Val x1434 = (x802[505]._super._super * Val(512));
  Val x1435 = (x802[506]._super._super * Val(1024));
  Val x1436 = (x802[507]._super._super * Val(2048));
  Val x1437 = (x802[508]._super._super * Val(4096));
  Val x1438 = (x802[509]._super._super * Val(8192));
  Val x1439 = (x802[510]._super._super * Val(16384));
  Val x1440 = (x802[511]._super._super * Val(32768));
  Val x1441 = (x802[496]._super._super + x1426);
  Val x1442 = (((x1441 + x1427) + x1428) + x1429);
  Val x1443 = (((x1442 + x1430) + x1431) + x1432);
  Val x1444 = (((x1443 + x1433) + x1434) + x1435);
  Val x1445 = (((x1444 + x1436) + x1437) + x1438);
  Val x1446 = (x802[513]._super._super * Val(2));
  Val x1447 = (x802[514]._super._super * Val(4));
  Val x1448 = (x802[515]._super._super * Val(8));
  Val x1449 = (x802[516]._super._super * Val(16));
  Val x1450 = (x802[517]._super._super * Val(32));
  Val x1451 = (x802[518]._super._super * Val(64));
  Val x1452 = (x802[519]._super._super * Val(128));
  Val x1453 = (x802[520]._super._super * Val(256));
  Val x1454 = (x802[521]._super._super * Val(512));
  Val x1455 = (x802[522]._super._super * Val(1024));
  Val x1456 = (x802[523]._super._super * Val(2048));
  Val x1457 = (x802[524]._super._super * Val(4096));
  Val x1458 = (x802[525]._super._super * Val(8192));
  Val x1459 = (x802[526]._super._super * Val(16384));
  Val x1460 = (x802[527]._super._super * Val(32768));
  Val x1461 = (x802[512]._super._super + x1446);
  Val x1462 = (((x1461 + x1447) + x1448) + x1449);
  Val x1463 = (((x1462 + x1450) + x1451) + x1452);
  Val x1464 = (((x1463 + x1453) + x1454) + x1455);
  Val x1465 = (((x1464 + x1456) + x1457) + x1458);
  Val x1466 = (x802[529]._super._super * Val(2));
  Val x1467 = (x802[530]._super._super * Val(4));
  Val x1468 = (x802[531]._super._super * Val(8));
  Val x1469 = (x802[532]._super._super * Val(16));
  Val x1470 = (x802[533]._super._super * Val(32));
  Val x1471 = (x802[534]._super._super * Val(64));
  Val x1472 = (x802[535]._super._super * Val(128));
  Val x1473 = (x802[536]._super._super * Val(256));
  Val x1474 = (x802[537]._super._super * Val(512));
  Val x1475 = (x802[538]._super._super * Val(1024));
  Val x1476 = (x802[539]._super._super * Val(2048));
  Val x1477 = (x802[540]._super._super * Val(4096));
  Val x1478 = (x802[541]._super._super * Val(8192));
  Val x1479 = (x802[542]._super._super * Val(16384));
  Val x1480 = (x802[543]._super._super * Val(32768));
  Val x1481 = (x802[528]._super._super + x1466);
  Val x1482 = (((x1481 + x1467) + x1468) + x1469);
  Val x1483 = (((x1482 + x1470) + x1471) + x1472);
  Val x1484 = (((x1483 + x1473) + x1474) + x1475);
  Val x1485 = (((x1484 + x1476) + x1477) + x1478);
  Val x1486 = (x802[545]._super._super * Val(2));
  Val x1487 = (x802[546]._super._super * Val(4));
  Val x1488 = (x802[547]._super._super * Val(8));
  Val x1489 = (x802[548]._super._super * Val(16));
  Val x1490 = (x802[549]._super._super * Val(32));
  Val x1491 = (x802[550]._super._super * Val(64));
  Val x1492 = (x802[551]._super._super * Val(128));
  Val x1493 = (x802[552]._super._super * Val(256));
  Val x1494 = (x802[553]._super._super * Val(512));
  Val x1495 = (x802[554]._super._super * Val(1024));
  Val x1496 = (x802[555]._super._super * Val(2048));
  Val x1497 = (x802[556]._super._super * Val(4096));
  Val x1498 = (x802[557]._super._super * Val(8192));
  Val x1499 = (x802[558]._super._super * Val(16384));
  Val x1500 = (x802[559]._super._super * Val(32768));
  Val x1501 = (x802[544]._super._super + x1486);
  Val x1502 = (((x1501 + x1487) + x1488) + x1489);
  Val x1503 = (((x1502 + x1490) + x1491) + x1492);
  Val x1504 = (((x1503 + x1493) + x1494) + x1495);
  Val x1505 = (((x1504 + x1496) + x1497) + x1498);
  Val x1506 = (x802[561]._super._super * Val(2));
  Val x1507 = (x802[562]._super._super * Val(4));
  Val x1508 = (x802[563]._super._super * Val(8));
  Val x1509 = (x802[564]._super._super * Val(16));
  Val x1510 = (x802[565]._super._super * Val(32));
  Val x1511 = (x802[566]._super._super * Val(64));
  Val x1512 = (x802[567]._super._super * Val(128));
  Val x1513 = (x802[568]._super._super * Val(256));
  Val x1514 = (x802[569]._super._super * Val(512));
  Val x1515 = (x802[570]._super._super * Val(1024));
  Val x1516 = (x802[571]._super._super * Val(2048));
  Val x1517 = (x802[572]._super._super * Val(4096));
  Val x1518 = (x802[573]._super._super * Val(8192));
  Val x1519 = (x802[574]._super._super * Val(16384));
  Val x1520 = (x802[575]._super._super * Val(32768));
  Val x1521 = (x802[560]._super._super + x1506);
  Val x1522 = (((x1521 + x1507) + x1508) + x1509);
  Val x1523 = (((x1522 + x1510) + x1511) + x1512);
  Val x1524 = (((x1523 + x1513) + x1514) + x1515);
  Val x1525 = (((x1524 + x1516) + x1517) + x1518);
  Val x1526 = (x802[577]._super._super * Val(2));
  Val x1527 = (x802[578]._super._super * Val(4));
  Val x1528 = (x802[579]._super._super * Val(8));
  Val x1529 = (x802[580]._super._super * Val(16));
  Val x1530 = (x802[581]._super._super * Val(32));
  Val x1531 = (x802[582]._super._super * Val(64));
  Val x1532 = (x802[583]._super._super * Val(128));
  Val x1533 = (x802[584]._super._super * Val(256));
  Val x1534 = (x802[585]._super._super * Val(512));
  Val x1535 = (x802[586]._super._super * Val(1024));
  Val x1536 = (x802[587]._super._super * Val(2048));
  Val x1537 = (x802[588]._super._super * Val(4096));
  Val x1538 = (x802[589]._super._super * Val(8192));
  Val x1539 = (x802[590]._super._super * Val(16384));
  Val x1540 = (x802[591]._super._super * Val(32768));
  Val x1541 = (x802[576]._super._super + x1526);
  Val x1542 = (((x1541 + x1527) + x1528) + x1529);
  Val x1543 = (((x1542 + x1530) + x1531) + x1532);
  Val x1544 = (((x1543 + x1533) + x1534) + x1535);
  Val x1545 = (((x1544 + x1536) + x1537) + x1538);
  Val x1546 = (x802[593]._super._super * Val(2));
  Val x1547 = (x802[594]._super._super * Val(4));
  Val x1548 = (x802[595]._super._super * Val(8));
  Val x1549 = (x802[596]._super._super * Val(16));
  Val x1550 = (x802[597]._super._super * Val(32));
  Val x1551 = (x802[598]._super._super * Val(64));
  Val x1552 = (x802[599]._super._super * Val(128));
  Val x1553 = (x802[600]._super._super * Val(256));
  Val x1554 = (x802[601]._super._super * Val(512));
  Val x1555 = (x802[602]._super._super * Val(1024));
  Val x1556 = (x802[603]._super._super * Val(2048));
  Val x1557 = (x802[604]._super._super * Val(4096));
  Val x1558 = (x802[605]._super._super * Val(8192));
  Val x1559 = (x802[606]._super._super * Val(16384));
  Val x1560 = (x802[607]._super._super * Val(32768));
  Val x1561 = (x802[592]._super._super + x1546);
  Val x1562 = (((x1561 + x1547) + x1548) + x1549);
  Val x1563 = (((x1562 + x1550) + x1551) + x1552);
  Val x1564 = (((x1563 + x1553) + x1554) + x1555);
  Val x1565 = (((x1564 + x1556) + x1557) + x1558);
  Val x1566 = (x802[609]._super._super * Val(2));
  Val x1567 = (x802[610]._super._super * Val(4));
  Val x1568 = (x802[611]._super._super * Val(8));
  Val x1569 = (x802[612]._super._super * Val(16));
  Val x1570 = (x802[613]._super._super * Val(32));
  Val x1571 = (x802[614]._super._super * Val(64));
  Val x1572 = (x802[615]._super._super * Val(128));
  Val x1573 = (x802[616]._super._super * Val(256));
  Val x1574 = (x802[617]._super._super * Val(512));
  Val x1575 = (x802[618]._super._super * Val(1024));
  Val x1576 = (x802[619]._super._super * Val(2048));
  Val x1577 = (x802[620]._super._super * Val(4096));
  Val x1578 = (x802[621]._super._super * Val(8192));
  Val x1579 = (x802[622]._super._super * Val(16384));
  Val x1580 = (x802[623]._super._super * Val(32768));
  Val x1581 = (x802[608]._super._super + x1566);
  Val x1582 = (((x1581 + x1567) + x1568) + x1569);
  Val x1583 = (((x1582 + x1570) + x1571) + x1572);
  Val x1584 = (((x1583 + x1573) + x1574) + x1575);
  Val x1585 = (((x1584 + x1576) + x1577) + x1578);
  Val x1586 = (x802[625]._super._super * Val(2));
  Val x1587 = (x802[626]._super._super * Val(4));
  Val x1588 = (x802[627]._super._super * Val(8));
  Val x1589 = (x802[628]._super._super * Val(16));
  Val x1590 = (x802[629]._super._super * Val(32));
  Val x1591 = (x802[630]._super._super * Val(64));
  Val x1592 = (x802[631]._super._super * Val(128));
  Val x1593 = (x802[632]._super._super * Val(256));
  Val x1594 = (x802[633]._super._super * Val(512));
  Val x1595 = (x802[634]._super._super * Val(1024));
  Val x1596 = (x802[635]._super._super * Val(2048));
  Val x1597 = (x802[636]._super._super * Val(4096));
  Val x1598 = (x802[637]._super._super * Val(8192));
  Val x1599 = (x802[638]._super._super * Val(16384));
  Val x1600 = (x802[639]._super._super * Val(32768));
  Val x1601 = (x802[624]._super._super + x1586);
  Val x1602 = (((x1601 + x1587) + x1588) + x1589);
  Val x1603 = (((x1602 + x1590) + x1591) + x1592);
  Val x1604 = (((x1603 + x1593) + x1594) + x1595);
  Val x1605 = (((x1604 + x1596) + x1597) + x1598);
  Val x1606 = (x802[641]._super._super * Val(2));
  Val x1607 = (x802[642]._super._super * Val(4));
  Val x1608 = (x802[643]._super._super * Val(8));
  Val x1609 = (x802[644]._super._super * Val(16));
  Val x1610 = (x802[645]._super._super * Val(32));
  Val x1611 = (x802[646]._super._super * Val(64));
  Val x1612 = (x802[647]._super._super * Val(128));
  Val x1613 = (x802[648]._super._super * Val(256));
  Val x1614 = (x802[649]._super._super * Val(512));
  Val x1615 = (x802[650]._super._super * Val(1024));
  Val x1616 = (x802[651]._super._super * Val(2048));
  Val x1617 = (x802[652]._super._super * Val(4096));
  Val x1618 = (x802[653]._super._super * Val(8192));
  Val x1619 = (x802[654]._super._super * Val(16384));
  Val x1620 = (x802[655]._super._super * Val(32768));
  Val x1621 = (x802[640]._super._super + x1606);
  Val x1622 = (((x1621 + x1607) + x1608) + x1609);
  Val x1623 = (((x1622 + x1610) + x1611) + x1612);
  Val x1624 = (((x1623 + x1613) + x1614) + x1615);
  Val x1625 = (((x1624 + x1616) + x1617) + x1618);
  Val x1626 = (x802[657]._super._super * Val(2));
  Val x1627 = (x802[658]._super._super * Val(4));
  Val x1628 = (x802[659]._super._super * Val(8));
  Val x1629 = (x802[660]._super._super * Val(16));
  Val x1630 = (x802[661]._super._super * Val(32));
  Val x1631 = (x802[662]._super._super * Val(64));
  Val x1632 = (x802[663]._super._super * Val(128));
  Val x1633 = (x802[664]._super._super * Val(256));
  Val x1634 = (x802[665]._super._super * Val(512));
  Val x1635 = (x802[666]._super._super * Val(1024));
  Val x1636 = (x802[667]._super._super * Val(2048));
  Val x1637 = (x802[668]._super._super * Val(4096));
  Val x1638 = (x802[669]._super._super * Val(8192));
  Val x1639 = (x802[670]._super._super * Val(16384));
  Val x1640 = (x802[671]._super._super * Val(32768));
  Val x1641 = (x802[656]._super._super + x1626);
  Val x1642 = (((x1641 + x1627) + x1628) + x1629);
  Val x1643 = (((x1642 + x1630) + x1631) + x1632);
  Val x1644 = (((x1643 + x1633) + x1634) + x1635);
  Val x1645 = (((x1644 + x1636) + x1637) + x1638);
  Val x1646 = (x802[673]._super._super * Val(2));
  Val x1647 = (x802[674]._super._super * Val(4));
  Val x1648 = (x802[675]._super._super * Val(8));
  Val x1649 = (x802[676]._super._super * Val(16));
  Val x1650 = (x802[677]._super._super * Val(32));
  Val x1651 = (x802[678]._super._super * Val(64));
  Val x1652 = (x802[679]._super._super * Val(128));
  Val x1653 = (x802[680]._super._super * Val(256));
  Val x1654 = (x802[681]._super._super * Val(512));
  Val x1655 = (x802[682]._super._super * Val(1024));
  Val x1656 = (x802[683]._super._super * Val(2048));
  Val x1657 = (x802[684]._super._super * Val(4096));
  Val x1658 = (x802[685]._super._super * Val(8192));
  Val x1659 = (x802[686]._super._super * Val(16384));
  Val x1660 = (x802[687]._super._super * Val(32768));
  Val x1661 = (x802[672]._super._super + x1646);
  Val x1662 = (((x1661 + x1647) + x1648) + x1649);
  Val x1663 = (((x1662 + x1650) + x1651) + x1652);
  Val x1664 = (((x1663 + x1653) + x1654) + x1655);
  Val x1665 = (((x1664 + x1656) + x1657) + x1658);
  Val x1666 = (x802[689]._super._super * Val(2));
  Val x1667 = (x802[690]._super._super * Val(4));
  Val x1668 = (x802[691]._super._super * Val(8));
  Val x1669 = (x802[692]._super._super * Val(16));
  Val x1670 = (x802[693]._super._super * Val(32));
  Val x1671 = (x802[694]._super._super * Val(64));
  Val x1672 = (x802[695]._super._super * Val(128));
  Val x1673 = (x802[696]._super._super * Val(256));
  Val x1674 = (x802[697]._super._super * Val(512));
  Val x1675 = (x802[698]._super._super * Val(1024));
  Val x1676 = (x802[699]._super._super * Val(2048));
  Val x1677 = (x802[700]._super._super * Val(4096));
  Val x1678 = (x802[701]._super._super * Val(8192));
  Val x1679 = (x802[702]._super._super * Val(16384));
  Val x1680 = (x802[703]._super._super * Val(32768));
  Val x1681 = (x802[688]._super._super + x1666);
  Val x1682 = (((x1681 + x1667) + x1668) + x1669);
  Val x1683 = (((x1682 + x1670) + x1671) + x1672);
  Val x1684 = (((x1683 + x1673) + x1674) + x1675);
  Val x1685 = (((x1684 + x1676) + x1677) + x1678);
  Val x1686 = (x802[705]._super._super * Val(2));
  Val x1687 = (x802[706]._super._super * Val(4));
  Val x1688 = (x802[707]._super._super * Val(8));
  Val x1689 = (x802[708]._super._super * Val(16));
  Val x1690 = (x802[709]._super._super * Val(32));
  Val x1691 = (x802[710]._super._super * Val(64));
  Val x1692 = (x802[711]._super._super * Val(128));
  Val x1693 = (x802[712]._super._super * Val(256));
  Val x1694 = (x802[713]._super._super * Val(512));
  Val x1695 = (x802[714]._super._super * Val(1024));
  Val x1696 = (x802[715]._super._super * Val(2048));
  Val x1697 = (x802[716]._super._super * Val(4096));
  Val x1698 = (x802[717]._super._super * Val(8192));
  Val x1699 = (x802[718]._super._super * Val(16384));
  Val x1700 = (x802[719]._super._super * Val(32768));
  Val x1701 = (x802[704]._super._super + x1686);
  Val x1702 = (((x1701 + x1687) + x1688) + x1689);
  Val x1703 = (((x1702 + x1690) + x1691) + x1692);
  Val x1704 = (((x1703 + x1693) + x1694) + x1695);
  Val x1705 = (((x1704 + x1696) + x1697) + x1698);
  Val x1706 = (x802[721]._super._super * Val(2));
  Val x1707 = (x802[722]._super._super * Val(4));
  Val x1708 = (x802[723]._super._super * Val(8));
  Val x1709 = (x802[724]._super._super * Val(16));
  Val x1710 = (x802[725]._super._super * Val(32));
  Val x1711 = (x802[726]._super._super * Val(64));
  Val x1712 = (x802[727]._super._super * Val(128));
  Val x1713 = (x802[728]._super._super * Val(256));
  Val x1714 = (x802[729]._super._super * Val(512));
  Val x1715 = (x802[730]._super._super * Val(1024));
  Val x1716 = (x802[731]._super._super * Val(2048));
  Val x1717 = (x802[732]._super._super * Val(4096));
  Val x1718 = (x802[733]._super._super * Val(8192));
  Val x1719 = (x802[734]._super._super * Val(16384));
  Val x1720 = (x802[735]._super._super * Val(32768));
  Val x1721 = (x802[720]._super._super + x1706);
  Val x1722 = (((x1721 + x1707) + x1708) + x1709);
  Val x1723 = (((x1722 + x1710) + x1711) + x1712);
  Val x1724 = (((x1723 + x1713) + x1714) + x1715);
  Val x1725 = (((x1724 + x1716) + x1717) + x1718);
  Val x1726 = (x802[737]._super._super * Val(2));
  Val x1727 = (x802[738]._super._super * Val(4));
  Val x1728 = (x802[739]._super._super * Val(8));
  Val x1729 = (x802[740]._super._super * Val(16));
  Val x1730 = (x802[741]._super._super * Val(32));
  Val x1731 = (x802[742]._super._super * Val(64));
  Val x1732 = (x802[743]._super._super * Val(128));
  Val x1733 = (x802[744]._super._super * Val(256));
  Val x1734 = (x802[745]._super._super * Val(512));
  Val x1735 = (x802[746]._super._super * Val(1024));
  Val x1736 = (x802[747]._super._super * Val(2048));
  Val x1737 = (x802[748]._super._super * Val(4096));
  Val x1738 = (x802[749]._super._super * Val(8192));
  Val x1739 = (x802[750]._super._super * Val(16384));
  Val x1740 = (x802[751]._super._super * Val(32768));
  Val x1741 = (x802[736]._super._super + x1726);
  Val x1742 = (((x1741 + x1727) + x1728) + x1729);
  Val x1743 = (((x1742 + x1730) + x1731) + x1732);
  Val x1744 = (((x1743 + x1733) + x1734) + x1735);
  Val x1745 = (((x1744 + x1736) + x1737) + x1738);
  Val x1746 = (x802[753]._super._super * Val(2));
  Val x1747 = (x802[754]._super._super * Val(4));
  Val x1748 = (x802[755]._super._super * Val(8));
  Val x1749 = (x802[756]._super._super * Val(16));
  Val x1750 = (x802[757]._super._super * Val(32));
  Val x1751 = (x802[758]._super._super * Val(64));
  Val x1752 = (x802[759]._super._super * Val(128));
  Val x1753 = (x802[760]._super._super * Val(256));
  Val x1754 = (x802[761]._super._super * Val(512));
  Val x1755 = (x802[762]._super._super * Val(1024));
  Val x1756 = (x802[763]._super._super * Val(2048));
  Val x1757 = (x802[764]._super._super * Val(4096));
  Val x1758 = (x802[765]._super._super * Val(8192));
  Val x1759 = (x802[766]._super._super * Val(16384));
  Val x1760 = (x802[767]._super._super * Val(32768));
  Val x1761 = (x802[752]._super._super + x1746);
  Val x1762 = (((x1761 + x1747) + x1748) + x1749);
  Val x1763 = (((x1762 + x1750) + x1751) + x1752);
  Val x1764 = (((x1763 + x1753) + x1754) + x1755);
  Val x1765 = (((x1764 + x1756) + x1757) + x1758);
  Val x1766 = (x802[769]._super._super * Val(2));
  Val x1767 = (x802[770]._super._super * Val(4));
  Val x1768 = (x802[771]._super._super * Val(8));
  Val x1769 = (x802[772]._super._super * Val(16));
  Val x1770 = (x802[773]._super._super * Val(32));
  Val x1771 = (x802[774]._super._super * Val(64));
  Val x1772 = (x802[775]._super._super * Val(128));
  Val x1773 = (x802[776]._super._super * Val(256));
  Val x1774 = (x802[777]._super._super * Val(512));
  Val x1775 = (x802[778]._super._super * Val(1024));
  Val x1776 = (x802[779]._super._super * Val(2048));
  Val x1777 = (x802[780]._super._super * Val(4096));
  Val x1778 = (x802[781]._super._super * Val(8192));
  Val x1779 = (x802[782]._super._super * Val(16384));
  Val x1780 = (x802[783]._super._super * Val(32768));
  Val x1781 = (x802[768]._super._super + x1766);
  Val x1782 = (((x1781 + x1767) + x1768) + x1769);
  Val x1783 = (((x1782 + x1770) + x1771) + x1772);
  Val x1784 = (((x1783 + x1773) + x1774) + x1775);
  Val x1785 = (((x1784 + x1776) + x1777) + x1778);
  Val x1786 = (x802[785]._super._super * Val(2));
  Val x1787 = (x802[786]._super._super * Val(4));
  Val x1788 = (x802[787]._super._super * Val(8));
  Val x1789 = (x802[788]._super._super * Val(16));
  Val x1790 = (x802[789]._super._super * Val(32));
  Val x1791 = (x802[790]._super._super * Val(64));
  Val x1792 = (x802[791]._super._super * Val(128));
  Val x1793 = (x802[792]._super._super * Val(256));
  Val x1794 = (x802[793]._super._super * Val(512));
  Val x1795 = (x802[794]._super._super * Val(1024));
  Val x1796 = (x802[795]._super._super * Val(2048));
  Val x1797 = (x802[796]._super._super * Val(4096));
  Val x1798 = (x802[797]._super._super * Val(8192));
  Val x1799 = (x802[798]._super._super * Val(16384));
  Val x1800 = (x802[799]._super._super * Val(32768));
  Val x1801 = (x802[784]._super._super + x1786);
  Val x1802 = (((x1801 + x1787) + x1788) + x1789);
  Val x1803 = (((x1802 + x1790) + x1791) + x1792);
  Val x1804 = (((x1803 + x1793) + x1794) + x1795);
  Val x1805 = (((x1804 + x1796) + x1797) + x1798);
  // EqArr(zirgen/circuit/keccak2/arr.zir:33)
  Val x1806 = (((x825 + x819) + x820) - arg0[0]);
  EQZ(x1806,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1807 = (((x845 + x839) + x840) - arg0[1]);
  EQZ(x1807,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1808 = (((x865 + x859) + x860) - arg0[2]);
  EQZ(x1808,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1809 = (((x885 + x879) + x880) - arg0[3]);
  EQZ(x1809,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1810 = (((x905 + x899) + x900) - arg0[4]);
  EQZ(x1810,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1811 = (((x925 + x919) + x920) - arg0[5]);
  EQZ(x1811,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1812 = (((x945 + x939) + x940) - arg0[6]);
  EQZ(x1812,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1813 = (((x965 + x959) + x960) - arg0[7]);
  EQZ(x1813,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1814 = (((x985 + x979) + x980) - arg0[8]);
  EQZ(x1814,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1815 = (((x1005 + x999) + x1000) - arg0[9]);
  EQZ(x1815,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1816 = (((x1025 + x1019) + x1020) - arg0[10]);
  EQZ(x1816,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1817 = (((x1045 + x1039) + x1040) - arg0[11]);
  EQZ(x1817,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1818 = (((x1065 + x1059) + x1060) - arg0[12]);
  EQZ(x1818,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1819 = (((x1085 + x1079) + x1080) - arg0[13]);
  EQZ(x1819,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1820 = (((x1105 + x1099) + x1100) - arg0[14]);
  EQZ(x1820,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1821 = (((x1125 + x1119) + x1120) - arg0[15]);
  EQZ(x1821,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1822 = (((x1145 + x1139) + x1140) - arg0[16]);
  EQZ(x1822,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1823 = (((x1165 + x1159) + x1160) - arg0[17]);
  EQZ(x1823,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1824 = (((x1185 + x1179) + x1180) - arg0[18]);
  EQZ(x1824,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1825 = (((x1205 + x1199) + x1200) - arg0[19]);
  EQZ(x1825,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1826 = (((x1225 + x1219) + x1220) - arg0[20]);
  EQZ(x1826,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1827 = (((x1245 + x1239) + x1240) - arg0[21]);
  EQZ(x1827,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1828 = (((x1265 + x1259) + x1260) - arg0[22]);
  EQZ(x1828,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1829 = (((x1285 + x1279) + x1280) - arg0[23]);
  EQZ(x1829,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1830 = (((x1305 + x1299) + x1300) - arg0[24]);
  EQZ(x1830,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1831 = (((x1325 + x1319) + x1320) - arg0[25]);
  EQZ(x1831,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1832 = (((x1345 + x1339) + x1340) - arg0[26]);
  EQZ(x1832,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1833 = (((x1365 + x1359) + x1360) - arg0[27]);
  EQZ(x1833,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1834 = (((x1385 + x1379) + x1380) - arg0[28]);
  EQZ(x1834,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1835 = (((x1405 + x1399) + x1400) - arg0[29]);
  EQZ(x1835,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1836 = (((x1425 + x1419) + x1420) - arg0[30]);
  EQZ(x1836,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1837 = (((x1445 + x1439) + x1440) - arg0[31]);
  EQZ(x1837,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1838 = (((x1465 + x1459) + x1460) - arg0[32]);
  EQZ(x1838,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1839 = (((x1485 + x1479) + x1480) - arg0[33]);
  EQZ(x1839,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1840 = (((x1505 + x1499) + x1500) - arg0[34]);
  EQZ(x1840,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1841 = (((x1525 + x1519) + x1520) - arg0[35]);
  EQZ(x1841,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1842 = (((x1545 + x1539) + x1540) - arg0[36]);
  EQZ(x1842,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1843 = (((x1565 + x1559) + x1560) - arg0[37]);
  EQZ(x1843,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1844 = (((x1585 + x1579) + x1580) - arg0[38]);
  EQZ(x1844,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1845 = (((x1605 + x1599) + x1600) - arg0[39]);
  EQZ(x1845,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1846 = (((x1625 + x1619) + x1620) - arg0[40]);
  EQZ(x1846,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1847 = (((x1645 + x1639) + x1640) - arg0[41]);
  EQZ(x1847,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1848 = (((x1665 + x1659) + x1660) - arg0[42]);
  EQZ(x1848,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1849 = (((x1685 + x1679) + x1680) - arg0[43]);
  EQZ(x1849,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1850 = (((x1705 + x1699) + x1700) - arg0[44]);
  EQZ(x1850,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1851 = (((x1725 + x1719) + x1720) - arg0[45]);
  EQZ(x1851,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1852 = (((x1745 + x1739) + x1740) - arg0[46]);
  EQZ(x1852,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1853 = (((x1765 + x1759) + x1760) - arg0[47]);
  EQZ(x1853,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1854 = (((x1785 + x1779) + x1780) - arg0[48]);
  EQZ(x1854,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  Val x1855 = (((x1805 + x1799) + x1800) - arg0[49]);
  EQZ(x1855,
      "loc(callsite( EqArr ( zirgen/circuit/keccak2/arr.zir :33:11) at  UnpackReg ( "
      "zirgen/circuit/keccak2/pack.zir :49:14)))");
  return UnpackReg_800__16_Struct{._super = x802};
}
__device__ TopStateStruct exec_KeccakRound12_0_(ExecContext& ctx,
                                                TopStateStruct arg0,
                                                TopStateStruct arg1,
                                                TopStateStruct arg2,
                                                TopStateStruct arg3,
                                                BoundLayout<TopStateLayout> layout4) {
  // Log(<preamble>:22)
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:108)
  INVOKE_EXTERN(ctx, log, "KeccakRound12", std::initializer_list<Val>{});
  // BitXor(zirgen/circuit/keccak2/bits.zir:32)
  // ThetaP2(zirgen/circuit/keccak2/keccak.zir:25)
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:111)
  Val x5 = (arg0.bits[256]._super._super + arg0.bits[127]._super._super);
  Val x6 = (arg0.bits[256]._super._super * Val(2));
  Val x7 = (x5 - (x6 * arg0.bits[127]._super._super));
  Val x8 = (arg0.bits[257]._super._super + arg0.bits[64]._super._super);
  Val x9 = (arg0.bits[257]._super._super * Val(2));
  Val x10 = (x8 - (x9 * arg0.bits[64]._super._super));
  Val x11 = (arg0.bits[258]._super._super + arg0.bits[65]._super._super);
  Val x12 = (arg0.bits[258]._super._super * Val(2));
  Val x13 = (x11 - (x12 * arg0.bits[65]._super._super));
  Val x14 = (arg0.bits[259]._super._super + arg0.bits[66]._super._super);
  Val x15 = (arg0.bits[259]._super._super * Val(2));
  Val x16 = (x14 - (x15 * arg0.bits[66]._super._super));
  Val x17 = (arg0.bits[260]._super._super + arg0.bits[67]._super._super);
  Val x18 = (arg0.bits[260]._super._super * Val(2));
  Val x19 = (x17 - (x18 * arg0.bits[67]._super._super));
  Val x20 = (arg0.bits[261]._super._super + arg0.bits[68]._super._super);
  Val x21 = (arg0.bits[261]._super._super * Val(2));
  Val x22 = (x20 - (x21 * arg0.bits[68]._super._super));
  Val x23 = (arg0.bits[262]._super._super + arg0.bits[69]._super._super);
  Val x24 = (arg0.bits[262]._super._super * Val(2));
  Val x25 = (x23 - (x24 * arg0.bits[69]._super._super));
  Val x26 = (arg0.bits[263]._super._super + arg0.bits[70]._super._super);
  Val x27 = (arg0.bits[263]._super._super * Val(2));
  Val x28 = (x26 - (x27 * arg0.bits[70]._super._super));
  Val x29 = (arg0.bits[264]._super._super + arg0.bits[71]._super._super);
  Val x30 = (arg0.bits[264]._super._super * Val(2));
  Val x31 = (x29 - (x30 * arg0.bits[71]._super._super));
  Val x32 = (arg0.bits[265]._super._super + arg0.bits[72]._super._super);
  Val x33 = (arg0.bits[265]._super._super * Val(2));
  Val x34 = (x32 - (x33 * arg0.bits[72]._super._super));
  Val x35 = (arg0.bits[266]._super._super + arg0.bits[73]._super._super);
  Val x36 = (arg0.bits[266]._super._super * Val(2));
  Val x37 = (x35 - (x36 * arg0.bits[73]._super._super));
  Val x38 = (arg0.bits[267]._super._super + arg0.bits[74]._super._super);
  Val x39 = (arg0.bits[267]._super._super * Val(2));
  Val x40 = (x38 - (x39 * arg0.bits[74]._super._super));
  Val x41 = (arg0.bits[268]._super._super + arg0.bits[75]._super._super);
  Val x42 = (arg0.bits[268]._super._super * Val(2));
  Val x43 = (x41 - (x42 * arg0.bits[75]._super._super));
  Val x44 = (arg0.bits[269]._super._super + arg0.bits[76]._super._super);
  Val x45 = (arg0.bits[269]._super._super * Val(2));
  Val x46 = (x44 - (x45 * arg0.bits[76]._super._super));
  Val x47 = (arg0.bits[270]._super._super + arg0.bits[77]._super._super);
  Val x48 = (arg0.bits[270]._super._super * Val(2));
  Val x49 = (x47 - (x48 * arg0.bits[77]._super._super));
  Val x50 = (arg0.bits[271]._super._super + arg0.bits[78]._super._super);
  Val x51 = (arg0.bits[271]._super._super * Val(2));
  Val x52 = (x50 - (x51 * arg0.bits[78]._super._super));
  Val x53 = (arg0.bits[272]._super._super + arg0.bits[79]._super._super);
  Val x54 = (arg0.bits[272]._super._super * Val(2));
  Val x55 = (x53 - (x54 * arg0.bits[79]._super._super));
  Val x56 = (arg0.bits[273]._super._super + arg0.bits[80]._super._super);
  Val x57 = (arg0.bits[273]._super._super * Val(2));
  Val x58 = (x56 - (x57 * arg0.bits[80]._super._super));
  Val x59 = (arg0.bits[274]._super._super + arg0.bits[81]._super._super);
  Val x60 = (arg0.bits[274]._super._super * Val(2));
  Val x61 = (x59 - (x60 * arg0.bits[81]._super._super));
  Val x62 = (arg0.bits[275]._super._super + arg0.bits[82]._super._super);
  Val x63 = (arg0.bits[275]._super._super * Val(2));
  Val x64 = (x62 - (x63 * arg0.bits[82]._super._super));
  Val x65 = (arg0.bits[276]._super._super + arg0.bits[83]._super._super);
  Val x66 = (arg0.bits[276]._super._super * Val(2));
  Val x67 = (x65 - (x66 * arg0.bits[83]._super._super));
  Val x68 = (arg0.bits[277]._super._super + arg0.bits[84]._super._super);
  Val x69 = (arg0.bits[277]._super._super * Val(2));
  Val x70 = (x68 - (x69 * arg0.bits[84]._super._super));
  Val x71 = (arg0.bits[278]._super._super + arg0.bits[85]._super._super);
  Val x72 = (arg0.bits[278]._super._super * Val(2));
  Val x73 = (x71 - (x72 * arg0.bits[85]._super._super));
  Val x74 = (arg0.bits[279]._super._super + arg0.bits[86]._super._super);
  Val x75 = (arg0.bits[279]._super._super * Val(2));
  Val x76 = (x74 - (x75 * arg0.bits[86]._super._super));
  Val x77 = (arg0.bits[280]._super._super + arg0.bits[87]._super._super);
  Val x78 = (arg0.bits[280]._super._super * Val(2));
  Val x79 = (x77 - (x78 * arg0.bits[87]._super._super));
  Val x80 = (arg0.bits[281]._super._super + arg0.bits[88]._super._super);
  Val x81 = (arg0.bits[281]._super._super * Val(2));
  Val x82 = (x80 - (x81 * arg0.bits[88]._super._super));
  Val x83 = (arg0.bits[282]._super._super + arg0.bits[89]._super._super);
  Val x84 = (arg0.bits[282]._super._super * Val(2));
  Val x85 = (x83 - (x84 * arg0.bits[89]._super._super));
  Val x86 = (arg0.bits[283]._super._super + arg0.bits[90]._super._super);
  Val x87 = (arg0.bits[283]._super._super * Val(2));
  Val x88 = (x86 - (x87 * arg0.bits[90]._super._super));
  Val x89 = (arg0.bits[284]._super._super + arg0.bits[91]._super._super);
  Val x90 = (arg0.bits[284]._super._super * Val(2));
  Val x91 = (x89 - (x90 * arg0.bits[91]._super._super));
  Val x92 = (arg0.bits[285]._super._super + arg0.bits[92]._super._super);
  Val x93 = (arg0.bits[285]._super._super * Val(2));
  Val x94 = (x92 - (x93 * arg0.bits[92]._super._super));
  Val x95 = (arg0.bits[286]._super._super + arg0.bits[93]._super._super);
  Val x96 = (arg0.bits[286]._super._super * Val(2));
  Val x97 = (x95 - (x96 * arg0.bits[93]._super._super));
  Val x98 = (arg0.bits[287]._super._super + arg0.bits[94]._super._super);
  Val x99 = (arg0.bits[287]._super._super * Val(2));
  Val x100 = (x98 - (x99 * arg0.bits[94]._super._super));
  Val x101 = (arg0.bits[288]._super._super + arg0.bits[95]._super._super);
  Val x102 = (arg0.bits[288]._super._super * Val(2));
  Val x103 = (x101 - (x102 * arg0.bits[95]._super._super));
  Val x104 = (arg0.bits[289]._super._super + arg0.bits[96]._super._super);
  Val x105 = (arg0.bits[289]._super._super * Val(2));
  Val x106 = (x104 - (x105 * arg0.bits[96]._super._super));
  Val x107 = (arg0.bits[290]._super._super + arg0.bits[97]._super._super);
  Val x108 = (arg0.bits[290]._super._super * Val(2));
  Val x109 = (x107 - (x108 * arg0.bits[97]._super._super));
  Val x110 = (arg0.bits[291]._super._super + arg0.bits[98]._super._super);
  Val x111 = (arg0.bits[291]._super._super * Val(2));
  Val x112 = (x110 - (x111 * arg0.bits[98]._super._super));
  Val x113 = (arg0.bits[292]._super._super + arg0.bits[99]._super._super);
  Val x114 = (arg0.bits[292]._super._super * Val(2));
  Val x115 = (x113 - (x114 * arg0.bits[99]._super._super));
  Val x116 = (arg0.bits[293]._super._super + arg0.bits[100]._super._super);
  Val x117 = (arg0.bits[293]._super._super * Val(2));
  Val x118 = (x116 - (x117 * arg0.bits[100]._super._super));
  Val x119 = (arg0.bits[294]._super._super + arg0.bits[101]._super._super);
  Val x120 = (arg0.bits[294]._super._super * Val(2));
  Val x121 = (x119 - (x120 * arg0.bits[101]._super._super));
  Val x122 = (arg0.bits[295]._super._super + arg0.bits[102]._super._super);
  Val x123 = (arg0.bits[295]._super._super * Val(2));
  Val x124 = (x122 - (x123 * arg0.bits[102]._super._super));
  Val x125 = (arg0.bits[296]._super._super + arg0.bits[103]._super._super);
  Val x126 = (arg0.bits[296]._super._super * Val(2));
  Val x127 = (x125 - (x126 * arg0.bits[103]._super._super));
  Val x128 = (arg0.bits[297]._super._super + arg0.bits[104]._super._super);
  Val x129 = (arg0.bits[297]._super._super * Val(2));
  Val x130 = (x128 - (x129 * arg0.bits[104]._super._super));
  Val x131 = (arg0.bits[298]._super._super + arg0.bits[105]._super._super);
  Val x132 = (arg0.bits[298]._super._super * Val(2));
  Val x133 = (x131 - (x132 * arg0.bits[105]._super._super));
  Val x134 = (arg0.bits[299]._super._super + arg0.bits[106]._super._super);
  Val x135 = (arg0.bits[299]._super._super * Val(2));
  Val x136 = (x134 - (x135 * arg0.bits[106]._super._super));
  Val x137 = (arg0.bits[300]._super._super + arg0.bits[107]._super._super);
  Val x138 = (arg0.bits[300]._super._super * Val(2));
  Val x139 = (x137 - (x138 * arg0.bits[107]._super._super));
  Val x140 = (arg0.bits[301]._super._super + arg0.bits[108]._super._super);
  Val x141 = (arg0.bits[301]._super._super * Val(2));
  Val x142 = (x140 - (x141 * arg0.bits[108]._super._super));
  Val x143 = (arg0.bits[302]._super._super + arg0.bits[109]._super._super);
  Val x144 = (arg0.bits[302]._super._super * Val(2));
  Val x145 = (x143 - (x144 * arg0.bits[109]._super._super));
  Val x146 = (arg0.bits[303]._super._super + arg0.bits[110]._super._super);
  Val x147 = (arg0.bits[303]._super._super * Val(2));
  Val x148 = (x146 - (x147 * arg0.bits[110]._super._super));
  Val x149 = (arg0.bits[304]._super._super + arg0.bits[111]._super._super);
  Val x150 = (arg0.bits[304]._super._super * Val(2));
  Val x151 = (x149 - (x150 * arg0.bits[111]._super._super));
  Val x152 = (arg0.bits[305]._super._super + arg0.bits[112]._super._super);
  Val x153 = (arg0.bits[305]._super._super * Val(2));
  Val x154 = (x152 - (x153 * arg0.bits[112]._super._super));
  Val x155 = (arg0.bits[306]._super._super + arg0.bits[113]._super._super);
  Val x156 = (arg0.bits[306]._super._super * Val(2));
  Val x157 = (x155 - (x156 * arg0.bits[113]._super._super));
  Val x158 = (arg0.bits[307]._super._super + arg0.bits[114]._super._super);
  Val x159 = (arg0.bits[307]._super._super * Val(2));
  Val x160 = (x158 - (x159 * arg0.bits[114]._super._super));
  Val x161 = (arg0.bits[308]._super._super + arg0.bits[115]._super._super);
  Val x162 = (arg0.bits[308]._super._super * Val(2));
  Val x163 = (x161 - (x162 * arg0.bits[115]._super._super));
  Val x164 = (arg0.bits[309]._super._super + arg0.bits[116]._super._super);
  Val x165 = (arg0.bits[309]._super._super * Val(2));
  Val x166 = (x164 - (x165 * arg0.bits[116]._super._super));
  Val x167 = (arg0.bits[310]._super._super + arg0.bits[117]._super._super);
  Val x168 = (arg0.bits[310]._super._super * Val(2));
  Val x169 = (x167 - (x168 * arg0.bits[117]._super._super));
  Val x170 = (arg0.bits[311]._super._super + arg0.bits[118]._super._super);
  Val x171 = (arg0.bits[311]._super._super * Val(2));
  Val x172 = (x170 - (x171 * arg0.bits[118]._super._super));
  Val x173 = (arg0.bits[312]._super._super + arg0.bits[119]._super._super);
  Val x174 = (arg0.bits[312]._super._super * Val(2));
  Val x175 = (x173 - (x174 * arg0.bits[119]._super._super));
  Val x176 = (arg0.bits[313]._super._super + arg0.bits[120]._super._super);
  Val x177 = (arg0.bits[313]._super._super * Val(2));
  Val x178 = (x176 - (x177 * arg0.bits[120]._super._super));
  Val x179 = (arg0.bits[314]._super._super + arg0.bits[121]._super._super);
  Val x180 = (arg0.bits[314]._super._super * Val(2));
  Val x181 = (x179 - (x180 * arg0.bits[121]._super._super));
  Val x182 = (arg0.bits[315]._super._super + arg0.bits[122]._super._super);
  Val x183 = (arg0.bits[315]._super._super * Val(2));
  Val x184 = (x182 - (x183 * arg0.bits[122]._super._super));
  Val x185 = (arg0.bits[316]._super._super + arg0.bits[123]._super._super);
  Val x186 = (arg0.bits[316]._super._super * Val(2));
  Val x187 = (x185 - (x186 * arg0.bits[123]._super._super));
  Val x188 = (arg0.bits[317]._super._super + arg0.bits[124]._super._super);
  Val x189 = (arg0.bits[317]._super._super * Val(2));
  Val x190 = (x188 - (x189 * arg0.bits[124]._super._super));
  Val x191 = (arg0.bits[318]._super._super + arg0.bits[125]._super._super);
  Val x192 = (arg0.bits[318]._super._super * Val(2));
  Val x193 = (x191 - (x192 * arg0.bits[125]._super._super));
  Val x194 = (arg0.bits[319]._super._super + arg0.bits[126]._super._super);
  Val x195 = (arg0.bits[319]._super._super * Val(2));
  Val x196 = (x194 - (x195 * arg0.bits[126]._super._super));
  Val x197 = (arg0.bits[0]._super._super + arg0.bits[191]._super._super);
  Val x198 = (arg0.bits[0]._super._super * Val(2));
  Val x199 = (x197 - (x198 * arg0.bits[191]._super._super));
  Val x200 = (arg0.bits[1]._super._super + arg0.bits[128]._super._super);
  Val x201 = (arg0.bits[1]._super._super * Val(2));
  Val x202 = (x200 - (x201 * arg0.bits[128]._super._super));
  Val x203 = (arg0.bits[2]._super._super + arg0.bits[129]._super._super);
  Val x204 = (arg0.bits[2]._super._super * Val(2));
  Val x205 = (x203 - (x204 * arg0.bits[129]._super._super));
  Val x206 = (arg0.bits[3]._super._super + arg0.bits[130]._super._super);
  Val x207 = (arg0.bits[3]._super._super * Val(2));
  Val x208 = (x206 - (x207 * arg0.bits[130]._super._super));
  Val x209 = (arg0.bits[4]._super._super + arg0.bits[131]._super._super);
  Val x210 = (arg0.bits[4]._super._super * Val(2));
  Val x211 = (x209 - (x210 * arg0.bits[131]._super._super));
  Val x212 = (arg0.bits[5]._super._super + arg0.bits[132]._super._super);
  Val x213 = (arg0.bits[5]._super._super * Val(2));
  Val x214 = (x212 - (x213 * arg0.bits[132]._super._super));
  Val x215 = (arg0.bits[6]._super._super + arg0.bits[133]._super._super);
  Val x216 = (arg0.bits[6]._super._super * Val(2));
  Val x217 = (x215 - (x216 * arg0.bits[133]._super._super));
  Val x218 = (arg0.bits[7]._super._super + arg0.bits[134]._super._super);
  Val x219 = (arg0.bits[7]._super._super * Val(2));
  Val x220 = (x218 - (x219 * arg0.bits[134]._super._super));
  Val x221 = (arg0.bits[8]._super._super + arg0.bits[135]._super._super);
  Val x222 = (arg0.bits[8]._super._super * Val(2));
  Val x223 = (x221 - (x222 * arg0.bits[135]._super._super));
  Val x224 = (arg0.bits[9]._super._super + arg0.bits[136]._super._super);
  Val x225 = (arg0.bits[9]._super._super * Val(2));
  Val x226 = (x224 - (x225 * arg0.bits[136]._super._super));
  Val x227 = (arg0.bits[10]._super._super + arg0.bits[137]._super._super);
  Val x228 = (arg0.bits[10]._super._super * Val(2));
  Val x229 = (x227 - (x228 * arg0.bits[137]._super._super));
  Val x230 = (arg0.bits[11]._super._super + arg0.bits[138]._super._super);
  Val x231 = (arg0.bits[11]._super._super * Val(2));
  Val x232 = (x230 - (x231 * arg0.bits[138]._super._super));
  Val x233 = (arg0.bits[12]._super._super + arg0.bits[139]._super._super);
  Val x234 = (arg0.bits[12]._super._super * Val(2));
  Val x235 = (x233 - (x234 * arg0.bits[139]._super._super));
  Val x236 = (arg0.bits[13]._super._super + arg0.bits[140]._super._super);
  Val x237 = (arg0.bits[13]._super._super * Val(2));
  Val x238 = (x236 - (x237 * arg0.bits[140]._super._super));
  Val x239 = (arg0.bits[14]._super._super + arg0.bits[141]._super._super);
  Val x240 = (arg0.bits[14]._super._super * Val(2));
  Val x241 = (x239 - (x240 * arg0.bits[141]._super._super));
  Val x242 = (arg0.bits[15]._super._super + arg0.bits[142]._super._super);
  Val x243 = (arg0.bits[15]._super._super * Val(2));
  Val x244 = (x242 - (x243 * arg0.bits[142]._super._super));
  Val x245 = (arg0.bits[16]._super._super + arg0.bits[143]._super._super);
  Val x246 = (arg0.bits[16]._super._super * Val(2));
  Val x247 = (x245 - (x246 * arg0.bits[143]._super._super));
  Val x248 = (arg0.bits[17]._super._super + arg0.bits[144]._super._super);
  Val x249 = (arg0.bits[17]._super._super * Val(2));
  Val x250 = (x248 - (x249 * arg0.bits[144]._super._super));
  Val x251 = (arg0.bits[18]._super._super + arg0.bits[145]._super._super);
  Val x252 = (arg0.bits[18]._super._super * Val(2));
  Val x253 = (x251 - (x252 * arg0.bits[145]._super._super));
  Val x254 = (arg0.bits[19]._super._super + arg0.bits[146]._super._super);
  Val x255 = (arg0.bits[19]._super._super * Val(2));
  Val x256 = (x254 - (x255 * arg0.bits[146]._super._super));
  Val x257 = (arg0.bits[20]._super._super + arg0.bits[147]._super._super);
  Val x258 = (arg0.bits[20]._super._super * Val(2));
  Val x259 = (x257 - (x258 * arg0.bits[147]._super._super));
  Val x260 = (arg0.bits[21]._super._super + arg0.bits[148]._super._super);
  Val x261 = (arg0.bits[21]._super._super * Val(2));
  Val x262 = (x260 - (x261 * arg0.bits[148]._super._super));
  Val x263 = (arg0.bits[22]._super._super + arg0.bits[149]._super._super);
  Val x264 = (arg0.bits[22]._super._super * Val(2));
  Val x265 = (x263 - (x264 * arg0.bits[149]._super._super));
  Val x266 = (arg0.bits[23]._super._super + arg0.bits[150]._super._super);
  Val x267 = (arg0.bits[23]._super._super * Val(2));
  Val x268 = (x266 - (x267 * arg0.bits[150]._super._super));
  Val x269 = (arg0.bits[24]._super._super + arg0.bits[151]._super._super);
  Val x270 = (arg0.bits[24]._super._super * Val(2));
  Val x271 = (x269 - (x270 * arg0.bits[151]._super._super));
  Val x272 = (arg0.bits[25]._super._super + arg0.bits[152]._super._super);
  Val x273 = (arg0.bits[25]._super._super * Val(2));
  Val x274 = (x272 - (x273 * arg0.bits[152]._super._super));
  Val x275 = (arg0.bits[26]._super._super + arg0.bits[153]._super._super);
  Val x276 = (arg0.bits[26]._super._super * Val(2));
  Val x277 = (x275 - (x276 * arg0.bits[153]._super._super));
  Val x278 = (arg0.bits[27]._super._super + arg0.bits[154]._super._super);
  Val x279 = (arg0.bits[27]._super._super * Val(2));
  Val x280 = (x278 - (x279 * arg0.bits[154]._super._super));
  Val x281 = (arg0.bits[28]._super._super + arg0.bits[155]._super._super);
  Val x282 = (arg0.bits[28]._super._super * Val(2));
  Val x283 = (x281 - (x282 * arg0.bits[155]._super._super));
  Val x284 = (arg0.bits[29]._super._super + arg0.bits[156]._super._super);
  Val x285 = (arg0.bits[29]._super._super * Val(2));
  Val x286 = (x284 - (x285 * arg0.bits[156]._super._super));
  Val x287 = (arg0.bits[30]._super._super + arg0.bits[157]._super._super);
  Val x288 = (arg0.bits[30]._super._super * Val(2));
  Val x289 = (x287 - (x288 * arg0.bits[157]._super._super));
  Val x290 = (arg0.bits[31]._super._super + arg0.bits[158]._super._super);
  Val x291 = (arg0.bits[31]._super._super * Val(2));
  Val x292 = (x290 - (x291 * arg0.bits[158]._super._super));
  Val x293 = (arg0.bits[32]._super._super + arg0.bits[159]._super._super);
  Val x294 = (arg0.bits[32]._super._super * Val(2));
  Val x295 = (x293 - (x294 * arg0.bits[159]._super._super));
  Val x296 = (arg0.bits[33]._super._super + arg0.bits[160]._super._super);
  Val x297 = (arg0.bits[33]._super._super * Val(2));
  Val x298 = (x296 - (x297 * arg0.bits[160]._super._super));
  Val x299 = (arg0.bits[34]._super._super + arg0.bits[161]._super._super);
  Val x300 = (arg0.bits[34]._super._super * Val(2));
  Val x301 = (x299 - (x300 * arg0.bits[161]._super._super));
  Val x302 = (arg0.bits[35]._super._super + arg0.bits[162]._super._super);
  Val x303 = (arg0.bits[35]._super._super * Val(2));
  Val x304 = (x302 - (x303 * arg0.bits[162]._super._super));
  Val x305 = (arg0.bits[36]._super._super + arg0.bits[163]._super._super);
  Val x306 = (arg0.bits[36]._super._super * Val(2));
  Val x307 = (x305 - (x306 * arg0.bits[163]._super._super));
  Val x308 = (arg0.bits[37]._super._super + arg0.bits[164]._super._super);
  Val x309 = (arg0.bits[37]._super._super * Val(2));
  Val x310 = (x308 - (x309 * arg0.bits[164]._super._super));
  Val x311 = (arg0.bits[38]._super._super + arg0.bits[165]._super._super);
  Val x312 = (arg0.bits[38]._super._super * Val(2));
  Val x313 = (x311 - (x312 * arg0.bits[165]._super._super));
  Val x314 = (arg0.bits[39]._super._super + arg0.bits[166]._super._super);
  Val x315 = (arg0.bits[39]._super._super * Val(2));
  Val x316 = (x314 - (x315 * arg0.bits[166]._super._super));
  Val x317 = (arg0.bits[40]._super._super + arg0.bits[167]._super._super);
  Val x318 = (arg0.bits[40]._super._super * Val(2));
  Val x319 = (x317 - (x318 * arg0.bits[167]._super._super));
  Val x320 = (arg0.bits[41]._super._super + arg0.bits[168]._super._super);
  Val x321 = (arg0.bits[41]._super._super * Val(2));
  Val x322 = (x320 - (x321 * arg0.bits[168]._super._super));
  Val x323 = (arg0.bits[42]._super._super + arg0.bits[169]._super._super);
  Val x324 = (arg0.bits[42]._super._super * Val(2));
  Val x325 = (x323 - (x324 * arg0.bits[169]._super._super));
  Val x326 = (arg0.bits[43]._super._super + arg0.bits[170]._super._super);
  Val x327 = (arg0.bits[43]._super._super * Val(2));
  Val x328 = (x326 - (x327 * arg0.bits[170]._super._super));
  Val x329 = (arg0.bits[44]._super._super + arg0.bits[171]._super._super);
  Val x330 = (arg0.bits[44]._super._super * Val(2));
  Val x331 = (x329 - (x330 * arg0.bits[171]._super._super));
  Val x332 = (arg0.bits[45]._super._super + arg0.bits[172]._super._super);
  Val x333 = (arg0.bits[45]._super._super * Val(2));
  Val x334 = (x332 - (x333 * arg0.bits[172]._super._super));
  Val x335 = (arg0.bits[46]._super._super + arg0.bits[173]._super._super);
  Val x336 = (arg0.bits[46]._super._super * Val(2));
  Val x337 = (x335 - (x336 * arg0.bits[173]._super._super));
  Val x338 = (arg0.bits[47]._super._super + arg0.bits[174]._super._super);
  Val x339 = (arg0.bits[47]._super._super * Val(2));
  Val x340 = (x338 - (x339 * arg0.bits[174]._super._super));
  Val x341 = (arg0.bits[48]._super._super + arg0.bits[175]._super._super);
  Val x342 = (arg0.bits[48]._super._super * Val(2));
  Val x343 = (x341 - (x342 * arg0.bits[175]._super._super));
  Val x344 = (arg0.bits[49]._super._super + arg0.bits[176]._super._super);
  Val x345 = (arg0.bits[49]._super._super * Val(2));
  Val x346 = (x344 - (x345 * arg0.bits[176]._super._super));
  Val x347 = (arg0.bits[50]._super._super + arg0.bits[177]._super._super);
  Val x348 = (arg0.bits[50]._super._super * Val(2));
  Val x349 = (x347 - (x348 * arg0.bits[177]._super._super));
  Val x350 = (arg0.bits[51]._super._super + arg0.bits[178]._super._super);
  Val x351 = (arg0.bits[51]._super._super * Val(2));
  Val x352 = (x350 - (x351 * arg0.bits[178]._super._super));
  Val x353 = (arg0.bits[54]._super._super + arg0.bits[181]._super._super);
  Val x354 = (arg0.bits[54]._super._super * Val(2));
  Val x355 = (x353 - (x354 * arg0.bits[181]._super._super));
  Val x356 = (arg0.bits[55]._super._super + arg0.bits[182]._super._super);
  Val x357 = (arg0.bits[55]._super._super * Val(2));
  Val x358 = (x356 - (x357 * arg0.bits[182]._super._super));
  Val x359 = (arg0.bits[56]._super._super + arg0.bits[183]._super._super);
  Val x360 = (arg0.bits[56]._super._super * Val(2));
  Val x361 = (x359 - (x360 * arg0.bits[183]._super._super));
  Val x362 = (arg0.bits[57]._super._super + arg0.bits[184]._super._super);
  Val x363 = (arg0.bits[57]._super._super * Val(2));
  Val x364 = (x362 - (x363 * arg0.bits[184]._super._super));
  Val x365 = (arg0.bits[58]._super._super + arg0.bits[185]._super._super);
  Val x366 = (arg0.bits[58]._super._super * Val(2));
  Val x367 = (x365 - (x366 * arg0.bits[185]._super._super));
  Val x368 = (arg0.bits[59]._super._super + arg0.bits[186]._super._super);
  Val x369 = (arg0.bits[59]._super._super * Val(2));
  Val x370 = (x368 - (x369 * arg0.bits[186]._super._super));
  Val x371 = (arg0.bits[60]._super._super + arg0.bits[187]._super._super);
  Val x372 = (arg0.bits[60]._super._super * Val(2));
  Val x373 = (x371 - (x372 * arg0.bits[187]._super._super));
  Val x374 = (arg0.bits[61]._super._super + arg0.bits[188]._super._super);
  Val x375 = (arg0.bits[61]._super._super * Val(2));
  Val x376 = (x374 - (x375 * arg0.bits[188]._super._super));
  Val x377 = (arg0.bits[62]._super._super + arg0.bits[189]._super._super);
  Val x378 = (arg0.bits[62]._super._super * Val(2));
  Val x379 = (x377 - (x378 * arg0.bits[189]._super._super));
  Val x380 = (arg0.bits[63]._super._super + arg0.bits[190]._super._super);
  Val x381 = (arg0.bits[63]._super._super * Val(2));
  Val x382 = (x380 - (x381 * arg0.bits[190]._super._super));
  Val x383 = (arg0.bits[64]._super._super + arg0.bits[255]._super._super);
  Val x384 = (arg0.bits[64]._super._super * Val(2));
  Val x385 = (x383 - (x384 * arg0.bits[255]._super._super));
  Val x386 = (arg0.bits[65]._super._super + arg0.bits[192]._super._super);
  Val x387 = (arg0.bits[65]._super._super * Val(2));
  Val x388 = (x386 - (x387 * arg0.bits[192]._super._super));
  Val x389 = (arg0.bits[66]._super._super + arg0.bits[193]._super._super);
  Val x390 = (arg0.bits[66]._super._super * Val(2));
  Val x391 = (x389 - (x390 * arg0.bits[193]._super._super));
  Val x392 = (arg0.bits[67]._super._super + arg0.bits[194]._super._super);
  Val x393 = (arg0.bits[67]._super._super * Val(2));
  Val x394 = (x392 - (x393 * arg0.bits[194]._super._super));
  Val x395 = (arg0.bits[68]._super._super + arg0.bits[195]._super._super);
  Val x396 = (arg0.bits[68]._super._super * Val(2));
  Val x397 = (x395 - (x396 * arg0.bits[195]._super._super));
  Val x398 = (arg0.bits[69]._super._super + arg0.bits[196]._super._super);
  Val x399 = (arg0.bits[69]._super._super * Val(2));
  Val x400 = (x398 - (x399 * arg0.bits[196]._super._super));
  Val x401 = (arg0.bits[70]._super._super + arg0.bits[197]._super._super);
  Val x402 = (arg0.bits[70]._super._super * Val(2));
  Val x403 = (x401 - (x402 * arg0.bits[197]._super._super));
  Val x404 = (arg0.bits[71]._super._super + arg0.bits[198]._super._super);
  Val x405 = (arg0.bits[71]._super._super * Val(2));
  Val x406 = (x404 - (x405 * arg0.bits[198]._super._super));
  Val x407 = (arg0.bits[72]._super._super + arg0.bits[199]._super._super);
  Val x408 = (arg0.bits[72]._super._super * Val(2));
  Val x409 = (x407 - (x408 * arg0.bits[199]._super._super));
  Val x410 = (arg0.bits[73]._super._super + arg0.bits[200]._super._super);
  Val x411 = (arg0.bits[73]._super._super * Val(2));
  Val x412 = (x410 - (x411 * arg0.bits[200]._super._super));
  Val x413 = (arg0.bits[74]._super._super + arg0.bits[201]._super._super);
  Val x414 = (arg0.bits[74]._super._super * Val(2));
  Val x415 = (x413 - (x414 * arg0.bits[201]._super._super));
  Val x416 = (arg0.bits[75]._super._super + arg0.bits[202]._super._super);
  Val x417 = (arg0.bits[75]._super._super * Val(2));
  Val x418 = (x416 - (x417 * arg0.bits[202]._super._super));
  Val x419 = (arg0.bits[76]._super._super + arg0.bits[203]._super._super);
  Val x420 = (arg0.bits[76]._super._super * Val(2));
  Val x421 = (x419 - (x420 * arg0.bits[203]._super._super));
  Val x422 = (arg0.bits[77]._super._super + arg0.bits[204]._super._super);
  Val x423 = (arg0.bits[77]._super._super * Val(2));
  Val x424 = (x422 - (x423 * arg0.bits[204]._super._super));
  Val x425 = (arg0.bits[78]._super._super + arg0.bits[205]._super._super);
  Val x426 = (arg0.bits[78]._super._super * Val(2));
  Val x427 = (x425 - (x426 * arg0.bits[205]._super._super));
  Val x428 = (arg0.bits[79]._super._super + arg0.bits[206]._super._super);
  Val x429 = (arg0.bits[79]._super._super * Val(2));
  Val x430 = (x428 - (x429 * arg0.bits[206]._super._super));
  Val x431 = (arg0.bits[80]._super._super + arg0.bits[207]._super._super);
  Val x432 = (arg0.bits[80]._super._super * Val(2));
  Val x433 = (x431 - (x432 * arg0.bits[207]._super._super));
  Val x434 = (arg0.bits[81]._super._super + arg0.bits[208]._super._super);
  Val x435 = (arg0.bits[81]._super._super * Val(2));
  Val x436 = (x434 - (x435 * arg0.bits[208]._super._super));
  Val x437 = (arg0.bits[82]._super._super + arg0.bits[209]._super._super);
  Val x438 = (arg0.bits[82]._super._super * Val(2));
  Val x439 = (x437 - (x438 * arg0.bits[209]._super._super));
  Val x440 = (arg0.bits[83]._super._super + arg0.bits[210]._super._super);
  Val x441 = (arg0.bits[83]._super._super * Val(2));
  Val x442 = (x440 - (x441 * arg0.bits[210]._super._super));
  Val x443 = (arg0.bits[84]._super._super + arg0.bits[211]._super._super);
  Val x444 = (arg0.bits[84]._super._super * Val(2));
  Val x445 = (x443 - (x444 * arg0.bits[211]._super._super));
  Val x446 = (arg0.bits[85]._super._super + arg0.bits[212]._super._super);
  Val x447 = (arg0.bits[85]._super._super * Val(2));
  Val x448 = (x446 - (x447 * arg0.bits[212]._super._super));
  Val x449 = (arg0.bits[86]._super._super + arg0.bits[213]._super._super);
  Val x450 = (arg0.bits[86]._super._super * Val(2));
  Val x451 = (x449 - (x450 * arg0.bits[213]._super._super));
  Val x452 = (arg0.bits[87]._super._super + arg0.bits[214]._super._super);
  Val x453 = (arg0.bits[87]._super._super * Val(2));
  Val x454 = (x452 - (x453 * arg0.bits[214]._super._super));
  Val x455 = (arg0.bits[88]._super._super + arg0.bits[215]._super._super);
  Val x456 = (arg0.bits[88]._super._super * Val(2));
  Val x457 = (x455 - (x456 * arg0.bits[215]._super._super));
  Val x458 = (arg0.bits[89]._super._super + arg0.bits[216]._super._super);
  Val x459 = (arg0.bits[89]._super._super * Val(2));
  Val x460 = (x458 - (x459 * arg0.bits[216]._super._super));
  Val x461 = (arg0.bits[90]._super._super + arg0.bits[217]._super._super);
  Val x462 = (arg0.bits[90]._super._super * Val(2));
  Val x463 = (x461 - (x462 * arg0.bits[217]._super._super));
  Val x464 = (arg0.bits[91]._super._super + arg0.bits[218]._super._super);
  Val x465 = (arg0.bits[91]._super._super * Val(2));
  Val x466 = (x464 - (x465 * arg0.bits[218]._super._super));
  Val x467 = (arg0.bits[92]._super._super + arg0.bits[219]._super._super);
  Val x468 = (arg0.bits[92]._super._super * Val(2));
  Val x469 = (x467 - (x468 * arg0.bits[219]._super._super));
  Val x470 = (arg0.bits[93]._super._super + arg0.bits[220]._super._super);
  Val x471 = (arg0.bits[93]._super._super * Val(2));
  Val x472 = (x470 - (x471 * arg0.bits[220]._super._super));
  Val x473 = (arg0.bits[94]._super._super + arg0.bits[221]._super._super);
  Val x474 = (arg0.bits[94]._super._super * Val(2));
  Val x475 = (x473 - (x474 * arg0.bits[221]._super._super));
  Val x476 = (arg0.bits[95]._super._super + arg0.bits[222]._super._super);
  Val x477 = (arg0.bits[95]._super._super * Val(2));
  Val x478 = (x476 - (x477 * arg0.bits[222]._super._super));
  Val x479 = (arg0.bits[96]._super._super + arg0.bits[223]._super._super);
  Val x480 = (arg0.bits[96]._super._super * Val(2));
  Val x481 = (x479 - (x480 * arg0.bits[223]._super._super));
  Val x482 = (arg0.bits[97]._super._super + arg0.bits[224]._super._super);
  Val x483 = (arg0.bits[97]._super._super * Val(2));
  Val x484 = (x482 - (x483 * arg0.bits[224]._super._super));
  Val x485 = (arg0.bits[98]._super._super + arg0.bits[225]._super._super);
  Val x486 = (arg0.bits[98]._super._super * Val(2));
  Val x487 = (x485 - (x486 * arg0.bits[225]._super._super));
  Val x488 = (arg0.bits[99]._super._super + arg0.bits[226]._super._super);
  Val x489 = (arg0.bits[99]._super._super * Val(2));
  Val x490 = (x488 - (x489 * arg0.bits[226]._super._super));
  Val x491 = (arg0.bits[100]._super._super + arg0.bits[227]._super._super);
  Val x492 = (arg0.bits[100]._super._super * Val(2));
  Val x493 = (x491 - (x492 * arg0.bits[227]._super._super));
  Val x494 = (arg0.bits[101]._super._super + arg0.bits[228]._super._super);
  Val x495 = (arg0.bits[101]._super._super * Val(2));
  Val x496 = (x494 - (x495 * arg0.bits[228]._super._super));
  Val x497 = (arg0.bits[102]._super._super + arg0.bits[229]._super._super);
  Val x498 = (arg0.bits[102]._super._super * Val(2));
  Val x499 = (x497 - (x498 * arg0.bits[229]._super._super));
  Val x500 = (arg0.bits[103]._super._super + arg0.bits[230]._super._super);
  Val x501 = (arg0.bits[103]._super._super * Val(2));
  Val x502 = (x500 - (x501 * arg0.bits[230]._super._super));
  Val x503 = (arg0.bits[104]._super._super + arg0.bits[231]._super._super);
  Val x504 = (arg0.bits[104]._super._super * Val(2));
  Val x505 = (x503 - (x504 * arg0.bits[231]._super._super));
  Val x506 = (arg0.bits[105]._super._super + arg0.bits[232]._super._super);
  Val x507 = (arg0.bits[105]._super._super * Val(2));
  Val x508 = (x506 - (x507 * arg0.bits[232]._super._super));
  Val x509 = (arg0.bits[106]._super._super + arg0.bits[233]._super._super);
  Val x510 = (arg0.bits[106]._super._super * Val(2));
  Val x511 = (x509 - (x510 * arg0.bits[233]._super._super));
  Val x512 = (arg0.bits[107]._super._super + arg0.bits[234]._super._super);
  Val x513 = (arg0.bits[107]._super._super * Val(2));
  Val x514 = (x512 - (x513 * arg0.bits[234]._super._super));
  Val x515 = (arg0.bits[108]._super._super + arg0.bits[235]._super._super);
  Val x516 = (arg0.bits[108]._super._super * Val(2));
  Val x517 = (x515 - (x516 * arg0.bits[235]._super._super));
  Val x518 = (arg0.bits[109]._super._super + arg0.bits[236]._super._super);
  Val x519 = (arg0.bits[109]._super._super * Val(2));
  Val x520 = (x518 - (x519 * arg0.bits[236]._super._super));
  Val x521 = (arg0.bits[110]._super._super + arg0.bits[237]._super._super);
  Val x522 = (arg0.bits[110]._super._super * Val(2));
  Val x523 = (x521 - (x522 * arg0.bits[237]._super._super));
  Val x524 = (arg0.bits[111]._super._super + arg0.bits[238]._super._super);
  Val x525 = (arg0.bits[111]._super._super * Val(2));
  Val x526 = (x524 - (x525 * arg0.bits[238]._super._super));
  Val x527 = (arg0.bits[112]._super._super + arg0.bits[239]._super._super);
  Val x528 = (arg0.bits[112]._super._super * Val(2));
  Val x529 = (x527 - (x528 * arg0.bits[239]._super._super));
  Val x530 = (arg0.bits[113]._super._super + arg0.bits[240]._super._super);
  Val x531 = (arg0.bits[113]._super._super * Val(2));
  Val x532 = (x530 - (x531 * arg0.bits[240]._super._super));
  Val x533 = (arg0.bits[114]._super._super + arg0.bits[241]._super._super);
  Val x534 = (arg0.bits[114]._super._super * Val(2));
  Val x535 = (x533 - (x534 * arg0.bits[241]._super._super));
  Val x536 = (arg0.bits[115]._super._super + arg0.bits[242]._super._super);
  Val x537 = (arg0.bits[115]._super._super * Val(2));
  Val x538 = (x536 - (x537 * arg0.bits[242]._super._super));
  Val x539 = (arg0.bits[116]._super._super + arg0.bits[243]._super._super);
  Val x540 = (arg0.bits[116]._super._super * Val(2));
  Val x541 = (x539 - (x540 * arg0.bits[243]._super._super));
  Val x542 = (arg0.bits[117]._super._super + arg0.bits[244]._super._super);
  Val x543 = (arg0.bits[117]._super._super * Val(2));
  Val x544 = (x542 - (x543 * arg0.bits[244]._super._super));
  Val x545 = (arg0.bits[118]._super._super + arg0.bits[245]._super._super);
  Val x546 = (arg0.bits[118]._super._super * Val(2));
  Val x547 = (x545 - (x546 * arg0.bits[245]._super._super));
  Val x548 = (arg0.bits[119]._super._super + arg0.bits[246]._super._super);
  Val x549 = (arg0.bits[119]._super._super * Val(2));
  Val x550 = (x548 - (x549 * arg0.bits[246]._super._super));
  Val x551 = (arg0.bits[120]._super._super + arg0.bits[247]._super._super);
  Val x552 = (arg0.bits[120]._super._super * Val(2));
  Val x553 = (x551 - (x552 * arg0.bits[247]._super._super));
  Val x554 = (arg0.bits[121]._super._super + arg0.bits[248]._super._super);
  Val x555 = (arg0.bits[121]._super._super * Val(2));
  Val x556 = (x554 - (x555 * arg0.bits[248]._super._super));
  Val x557 = (arg0.bits[122]._super._super + arg0.bits[249]._super._super);
  Val x558 = (arg0.bits[122]._super._super * Val(2));
  Val x559 = (x557 - (x558 * arg0.bits[249]._super._super));
  Val x560 = (arg0.bits[123]._super._super + arg0.bits[250]._super._super);
  Val x561 = (arg0.bits[123]._super._super * Val(2));
  Val x562 = (x560 - (x561 * arg0.bits[250]._super._super));
  Val x563 = (arg0.bits[124]._super._super + arg0.bits[251]._super._super);
  Val x564 = (arg0.bits[124]._super._super * Val(2));
  Val x565 = (x563 - (x564 * arg0.bits[251]._super._super));
  Val x566 = (arg0.bits[125]._super._super + arg0.bits[252]._super._super);
  Val x567 = (arg0.bits[125]._super._super * Val(2));
  Val x568 = (x566 - (x567 * arg0.bits[252]._super._super));
  Val x569 = (arg0.bits[126]._super._super + arg0.bits[253]._super._super);
  Val x570 = (arg0.bits[126]._super._super * Val(2));
  Val x571 = (x569 - (x570 * arg0.bits[253]._super._super));
  Val x572 = (arg0.bits[127]._super._super + arg0.bits[254]._super._super);
  Val x573 = (arg0.bits[127]._super._super * Val(2));
  Val x574 = (x572 - (x573 * arg0.bits[254]._super._super));
  Val x575 = (arg0.bits[128]._super._super + arg0.bits[319]._super._super);
  Val x576 = (arg0.bits[128]._super._super * Val(2));
  Val x577 = (x575 - (x576 * arg0.bits[319]._super._super));
  Val x578 = (arg0.bits[129]._super._super + arg0.bits[256]._super._super);
  Val x579 = (arg0.bits[129]._super._super * Val(2));
  Val x580 = (x578 - (x579 * arg0.bits[256]._super._super));
  Val x581 = (arg0.bits[130]._super._super + arg0.bits[257]._super._super);
  Val x582 = (arg0.bits[130]._super._super * Val(2));
  Val x583 = (x581 - (x582 * arg0.bits[257]._super._super));
  Val x584 = (arg0.bits[131]._super._super + arg0.bits[258]._super._super);
  Val x585 = (arg0.bits[131]._super._super * Val(2));
  Val x586 = (x584 - (x585 * arg0.bits[258]._super._super));
  Val x587 = (arg0.bits[132]._super._super + arg0.bits[259]._super._super);
  Val x588 = (arg0.bits[132]._super._super * Val(2));
  Val x589 = (x587 - (x588 * arg0.bits[259]._super._super));
  Val x590 = (arg0.bits[133]._super._super + arg0.bits[260]._super._super);
  Val x591 = (arg0.bits[133]._super._super * Val(2));
  Val x592 = (x590 - (x591 * arg0.bits[260]._super._super));
  Val x593 = (arg0.bits[134]._super._super + arg0.bits[261]._super._super);
  Val x594 = (arg0.bits[134]._super._super * Val(2));
  Val x595 = (x593 - (x594 * arg0.bits[261]._super._super));
  Val x596 = (arg0.bits[135]._super._super + arg0.bits[262]._super._super);
  Val x597 = (arg0.bits[135]._super._super * Val(2));
  Val x598 = (x596 - (x597 * arg0.bits[262]._super._super));
  Val x599 = (arg0.bits[136]._super._super + arg0.bits[263]._super._super);
  Val x600 = (arg0.bits[136]._super._super * Val(2));
  Val x601 = (x599 - (x600 * arg0.bits[263]._super._super));
  Val x602 = (arg0.bits[137]._super._super + arg0.bits[264]._super._super);
  Val x603 = (arg0.bits[137]._super._super * Val(2));
  Val x604 = (x602 - (x603 * arg0.bits[264]._super._super));
  Val x605 = (arg0.bits[138]._super._super + arg0.bits[265]._super._super);
  Val x606 = (arg0.bits[138]._super._super * Val(2));
  Val x607 = (x605 - (x606 * arg0.bits[265]._super._super));
  Val x608 = (arg0.bits[139]._super._super + arg0.bits[266]._super._super);
  Val x609 = (arg0.bits[139]._super._super * Val(2));
  Val x610 = (x608 - (x609 * arg0.bits[266]._super._super));
  Val x611 = (arg0.bits[140]._super._super + arg0.bits[267]._super._super);
  Val x612 = (arg0.bits[140]._super._super * Val(2));
  Val x613 = (x611 - (x612 * arg0.bits[267]._super._super));
  Val x614 = (arg0.bits[141]._super._super + arg0.bits[268]._super._super);
  Val x615 = (arg0.bits[141]._super._super * Val(2));
  Val x616 = (x614 - (x615 * arg0.bits[268]._super._super));
  Val x617 = (arg0.bits[142]._super._super + arg0.bits[269]._super._super);
  Val x618 = (arg0.bits[142]._super._super * Val(2));
  Val x619 = (x617 - (x618 * arg0.bits[269]._super._super));
  Val x620 = (arg0.bits[143]._super._super + arg0.bits[270]._super._super);
  Val x621 = (arg0.bits[143]._super._super * Val(2));
  Val x622 = (x620 - (x621 * arg0.bits[270]._super._super));
  Val x623 = (arg0.bits[144]._super._super + arg0.bits[271]._super._super);
  Val x624 = (arg0.bits[144]._super._super * Val(2));
  Val x625 = (x623 - (x624 * arg0.bits[271]._super._super));
  Val x626 = (arg0.bits[145]._super._super + arg0.bits[272]._super._super);
  Val x627 = (arg0.bits[145]._super._super * Val(2));
  Val x628 = (x626 - (x627 * arg0.bits[272]._super._super));
  Val x629 = (arg0.bits[146]._super._super + arg0.bits[273]._super._super);
  Val x630 = (arg0.bits[146]._super._super * Val(2));
  Val x631 = (x629 - (x630 * arg0.bits[273]._super._super));
  Val x632 = (arg0.bits[147]._super._super + arg0.bits[274]._super._super);
  Val x633 = (arg0.bits[147]._super._super * Val(2));
  Val x634 = (x632 - (x633 * arg0.bits[274]._super._super));
  Val x635 = (arg0.bits[148]._super._super + arg0.bits[275]._super._super);
  Val x636 = (arg0.bits[148]._super._super * Val(2));
  Val x637 = (x635 - (x636 * arg0.bits[275]._super._super));
  Val x638 = (arg0.bits[149]._super._super + arg0.bits[276]._super._super);
  Val x639 = (arg0.bits[149]._super._super * Val(2));
  Val x640 = (x638 - (x639 * arg0.bits[276]._super._super));
  Val x641 = (arg0.bits[150]._super._super + arg0.bits[277]._super._super);
  Val x642 = (arg0.bits[150]._super._super * Val(2));
  Val x643 = (x641 - (x642 * arg0.bits[277]._super._super));
  Val x644 = (arg0.bits[151]._super._super + arg0.bits[278]._super._super);
  Val x645 = (arg0.bits[151]._super._super * Val(2));
  Val x646 = (x644 - (x645 * arg0.bits[278]._super._super));
  Val x647 = (arg0.bits[152]._super._super + arg0.bits[279]._super._super);
  Val x648 = (arg0.bits[152]._super._super * Val(2));
  Val x649 = (x647 - (x648 * arg0.bits[279]._super._super));
  Val x650 = (arg0.bits[153]._super._super + arg0.bits[280]._super._super);
  Val x651 = (arg0.bits[153]._super._super * Val(2));
  Val x652 = (x650 - (x651 * arg0.bits[280]._super._super));
  Val x653 = (arg0.bits[154]._super._super + arg0.bits[281]._super._super);
  Val x654 = (arg0.bits[154]._super._super * Val(2));
  Val x655 = (x653 - (x654 * arg0.bits[281]._super._super));
  Val x656 = (arg0.bits[155]._super._super + arg0.bits[282]._super._super);
  Val x657 = (arg0.bits[155]._super._super * Val(2));
  Val x658 = (x656 - (x657 * arg0.bits[282]._super._super));
  Val x659 = (arg0.bits[156]._super._super + arg0.bits[283]._super._super);
  Val x660 = (arg0.bits[156]._super._super * Val(2));
  Val x661 = (x659 - (x660 * arg0.bits[283]._super._super));
  Val x662 = (arg0.bits[157]._super._super + arg0.bits[284]._super._super);
  Val x663 = (arg0.bits[157]._super._super * Val(2));
  Val x664 = (x662 - (x663 * arg0.bits[284]._super._super));
  Val x665 = (arg0.bits[158]._super._super + arg0.bits[285]._super._super);
  Val x666 = (arg0.bits[158]._super._super * Val(2));
  Val x667 = (x665 - (x666 * arg0.bits[285]._super._super));
  Val x668 = (arg0.bits[159]._super._super + arg0.bits[286]._super._super);
  Val x669 = (arg0.bits[159]._super._super * Val(2));
  Val x670 = (x668 - (x669 * arg0.bits[286]._super._super));
  Val x671 = (arg0.bits[160]._super._super + arg0.bits[287]._super._super);
  Val x672 = (arg0.bits[160]._super._super * Val(2));
  Val x673 = (x671 - (x672 * arg0.bits[287]._super._super));
  Val x674 = (arg0.bits[161]._super._super + arg0.bits[288]._super._super);
  Val x675 = (arg0.bits[161]._super._super * Val(2));
  Val x676 = (x674 - (x675 * arg0.bits[288]._super._super));
  Val x677 = (arg0.bits[162]._super._super + arg0.bits[289]._super._super);
  Val x678 = (arg0.bits[162]._super._super * Val(2));
  Val x679 = (x677 - (x678 * arg0.bits[289]._super._super));
  Val x680 = (arg0.bits[163]._super._super + arg0.bits[290]._super._super);
  Val x681 = (arg0.bits[163]._super._super * Val(2));
  Val x682 = (x680 - (x681 * arg0.bits[290]._super._super));
  Val x683 = (arg0.bits[164]._super._super + arg0.bits[291]._super._super);
  Val x684 = (arg0.bits[164]._super._super * Val(2));
  Val x685 = (x683 - (x684 * arg0.bits[291]._super._super));
  Val x686 = (arg0.bits[165]._super._super + arg0.bits[292]._super._super);
  Val x687 = (arg0.bits[165]._super._super * Val(2));
  Val x688 = (x686 - (x687 * arg0.bits[292]._super._super));
  Val x689 = (arg0.bits[166]._super._super + arg0.bits[293]._super._super);
  Val x690 = (arg0.bits[166]._super._super * Val(2));
  Val x691 = (x689 - (x690 * arg0.bits[293]._super._super));
  Val x692 = (arg0.bits[167]._super._super + arg0.bits[294]._super._super);
  Val x693 = (arg0.bits[167]._super._super * Val(2));
  Val x694 = (x692 - (x693 * arg0.bits[294]._super._super));
  Val x695 = (arg0.bits[168]._super._super + arg0.bits[295]._super._super);
  Val x696 = (arg0.bits[168]._super._super * Val(2));
  Val x697 = (x695 - (x696 * arg0.bits[295]._super._super));
  Val x698 = (arg0.bits[169]._super._super + arg0.bits[296]._super._super);
  Val x699 = (arg0.bits[169]._super._super * Val(2));
  Val x700 = (x698 - (x699 * arg0.bits[296]._super._super));
  Val x701 = (arg0.bits[170]._super._super + arg0.bits[297]._super._super);
  Val x702 = (arg0.bits[170]._super._super * Val(2));
  Val x703 = (x701 - (x702 * arg0.bits[297]._super._super));
  Val x704 = (arg0.bits[171]._super._super + arg0.bits[298]._super._super);
  Val x705 = (arg0.bits[171]._super._super * Val(2));
  Val x706 = (x704 - (x705 * arg0.bits[298]._super._super));
  Val x707 = (arg0.bits[172]._super._super + arg0.bits[299]._super._super);
  Val x708 = (arg0.bits[172]._super._super * Val(2));
  Val x709 = (x707 - (x708 * arg0.bits[299]._super._super));
  Val x710 = (arg0.bits[173]._super._super + arg0.bits[300]._super._super);
  Val x711 = (arg0.bits[173]._super._super * Val(2));
  Val x712 = (x710 - (x711 * arg0.bits[300]._super._super));
  Val x713 = (arg0.bits[174]._super._super + arg0.bits[301]._super._super);
  Val x714 = (arg0.bits[174]._super._super * Val(2));
  Val x715 = (x713 - (x714 * arg0.bits[301]._super._super));
  Val x716 = (arg0.bits[175]._super._super + arg0.bits[302]._super._super);
  Val x717 = (arg0.bits[175]._super._super * Val(2));
  Val x718 = (x716 - (x717 * arg0.bits[302]._super._super));
  Val x719 = (arg0.bits[176]._super._super + arg0.bits[303]._super._super);
  Val x720 = (arg0.bits[176]._super._super * Val(2));
  Val x721 = (x719 - (x720 * arg0.bits[303]._super._super));
  Val x722 = (arg0.bits[177]._super._super + arg0.bits[304]._super._super);
  Val x723 = (arg0.bits[177]._super._super * Val(2));
  Val x724 = (x722 - (x723 * arg0.bits[304]._super._super));
  Val x725 = (arg0.bits[178]._super._super + arg0.bits[305]._super._super);
  Val x726 = (arg0.bits[178]._super._super * Val(2));
  Val x727 = (x725 - (x726 * arg0.bits[305]._super._super));
  Val x728 = (arg0.bits[179]._super._super + arg0.bits[306]._super._super);
  Val x729 = (arg0.bits[179]._super._super * Val(2));
  Val x730 = (x728 - (x729 * arg0.bits[306]._super._super));
  Val x731 = (arg0.bits[180]._super._super + arg0.bits[307]._super._super);
  Val x732 = (arg0.bits[180]._super._super * Val(2));
  Val x733 = (x731 - (x732 * arg0.bits[307]._super._super));
  Val x734 = (arg0.bits[181]._super._super + arg0.bits[308]._super._super);
  Val x735 = (arg0.bits[181]._super._super * Val(2));
  Val x736 = (x734 - (x735 * arg0.bits[308]._super._super));
  Val x737 = (arg0.bits[182]._super._super + arg0.bits[309]._super._super);
  Val x738 = (arg0.bits[182]._super._super * Val(2));
  Val x739 = (x737 - (x738 * arg0.bits[309]._super._super));
  Val x740 = (arg0.bits[183]._super._super + arg0.bits[310]._super._super);
  Val x741 = (arg0.bits[183]._super._super * Val(2));
  Val x742 = (x740 - (x741 * arg0.bits[310]._super._super));
  Val x743 = (arg0.bits[184]._super._super + arg0.bits[311]._super._super);
  Val x744 = (arg0.bits[184]._super._super * Val(2));
  Val x745 = (x743 - (x744 * arg0.bits[311]._super._super));
  Val x746 = (arg0.bits[185]._super._super + arg0.bits[312]._super._super);
  Val x747 = (arg0.bits[185]._super._super * Val(2));
  Val x748 = (x746 - (x747 * arg0.bits[312]._super._super));
  Val x749 = (arg0.bits[186]._super._super + arg0.bits[313]._super._super);
  Val x750 = (arg0.bits[186]._super._super * Val(2));
  Val x751 = (x749 - (x750 * arg0.bits[313]._super._super));
  Val x752 = (arg0.bits[187]._super._super + arg0.bits[314]._super._super);
  Val x753 = (arg0.bits[187]._super._super * Val(2));
  Val x754 = (x752 - (x753 * arg0.bits[314]._super._super));
  Val x755 = (arg0.bits[188]._super._super + arg0.bits[315]._super._super);
  Val x756 = (arg0.bits[188]._super._super * Val(2));
  Val x757 = (x755 - (x756 * arg0.bits[315]._super._super));
  Val x758 = (arg0.bits[189]._super._super + arg0.bits[316]._super._super);
  Val x759 = (arg0.bits[189]._super._super * Val(2));
  Val x760 = (x758 - (x759 * arg0.bits[316]._super._super));
  Val x761 = (arg0.bits[190]._super._super + arg0.bits[317]._super._super);
  Val x762 = (arg0.bits[190]._super._super * Val(2));
  Val x763 = (x761 - (x762 * arg0.bits[317]._super._super));
  Val x764 = (arg0.bits[191]._super._super + arg0.bits[318]._super._super);
  Val x765 = (arg0.bits[191]._super._super * Val(2));
  Val x766 = (x764 - (x765 * arg0.bits[318]._super._super));
  Val x767 = (arg0.bits[192]._super._super + arg0.bits[63]._super._super);
  Val x768 = (arg0.bits[192]._super._super * Val(2));
  Val x769 = (x767 - (x768 * arg0.bits[63]._super._super));
  Val x770 = (arg0.bits[193]._super._super + arg0.bits[0]._super._super);
  Val x771 = (arg0.bits[193]._super._super * Val(2));
  Val x772 = (x770 - (x771 * arg0.bits[0]._super._super));
  Val x773 = (arg0.bits[194]._super._super + arg0.bits[1]._super._super);
  Val x774 = (arg0.bits[194]._super._super * Val(2));
  Val x775 = (x773 - (x774 * arg0.bits[1]._super._super));
  Val x776 = (arg0.bits[195]._super._super + arg0.bits[2]._super._super);
  Val x777 = (arg0.bits[195]._super._super * Val(2));
  Val x778 = (x776 - (x777 * arg0.bits[2]._super._super));
  Val x779 = (arg0.bits[196]._super._super + arg0.bits[3]._super._super);
  Val x780 = (arg0.bits[196]._super._super * Val(2));
  Val x781 = (x779 - (x780 * arg0.bits[3]._super._super));
  Val x782 = (arg0.bits[197]._super._super + arg0.bits[4]._super._super);
  Val x783 = (arg0.bits[197]._super._super * Val(2));
  Val x784 = (x782 - (x783 * arg0.bits[4]._super._super));
  Val x785 = (arg0.bits[198]._super._super + arg0.bits[5]._super._super);
  Val x786 = (arg0.bits[198]._super._super * Val(2));
  Val x787 = (x785 - (x786 * arg0.bits[5]._super._super));
  Val x788 = (arg0.bits[199]._super._super + arg0.bits[6]._super._super);
  Val x789 = (arg0.bits[199]._super._super * Val(2));
  Val x790 = (x788 - (x789 * arg0.bits[6]._super._super));
  Val x791 = (arg0.bits[200]._super._super + arg0.bits[7]._super._super);
  Val x792 = (arg0.bits[200]._super._super * Val(2));
  Val x793 = (x791 - (x792 * arg0.bits[7]._super._super));
  Val x794 = (arg0.bits[201]._super._super + arg0.bits[8]._super._super);
  Val x795 = (arg0.bits[201]._super._super * Val(2));
  Val x796 = (x794 - (x795 * arg0.bits[8]._super._super));
  Val x797 = (arg0.bits[202]._super._super + arg0.bits[9]._super._super);
  Val x798 = (arg0.bits[202]._super._super * Val(2));
  Val x799 = (x797 - (x798 * arg0.bits[9]._super._super));
  Val x800 = (arg0.bits[203]._super._super + arg0.bits[10]._super._super);
  Val x801 = (arg0.bits[203]._super._super * Val(2));
  Val x802 = (x800 - (x801 * arg0.bits[10]._super._super));
  Val x803 = (arg0.bits[204]._super._super + arg0.bits[11]._super._super);
  Val x804 = (arg0.bits[204]._super._super * Val(2));
  Val x805 = (x803 - (x804 * arg0.bits[11]._super._super));
  Val x806 = (arg0.bits[205]._super._super + arg0.bits[12]._super._super);
  Val x807 = (arg0.bits[205]._super._super * Val(2));
  Val x808 = (x806 - (x807 * arg0.bits[12]._super._super));
  Val x809 = (arg0.bits[206]._super._super + arg0.bits[13]._super._super);
  Val x810 = (arg0.bits[206]._super._super * Val(2));
  Val x811 = (x809 - (x810 * arg0.bits[13]._super._super));
  Val x812 = (arg0.bits[207]._super._super + arg0.bits[14]._super._super);
  Val x813 = (arg0.bits[207]._super._super * Val(2));
  Val x814 = (x812 - (x813 * arg0.bits[14]._super._super));
  Val x815 = (arg0.bits[208]._super._super + arg0.bits[15]._super._super);
  Val x816 = (arg0.bits[208]._super._super * Val(2));
  Val x817 = (x815 - (x816 * arg0.bits[15]._super._super));
  Val x818 = (arg0.bits[209]._super._super + arg0.bits[16]._super._super);
  Val x819 = (arg0.bits[209]._super._super * Val(2));
  Val x820 = (x818 - (x819 * arg0.bits[16]._super._super));
  Val x821 = (arg0.bits[210]._super._super + arg0.bits[17]._super._super);
  Val x822 = (arg0.bits[210]._super._super * Val(2));
  Val x823 = (x821 - (x822 * arg0.bits[17]._super._super));
  Val x824 = (arg0.bits[211]._super._super + arg0.bits[18]._super._super);
  Val x825 = (arg0.bits[211]._super._super * Val(2));
  Val x826 = (x824 - (x825 * arg0.bits[18]._super._super));
  Val x827 = (arg0.bits[212]._super._super + arg0.bits[19]._super._super);
  Val x828 = (arg0.bits[212]._super._super * Val(2));
  Val x829 = (x827 - (x828 * arg0.bits[19]._super._super));
  Val x830 = (arg0.bits[213]._super._super + arg0.bits[20]._super._super);
  Val x831 = (arg0.bits[213]._super._super * Val(2));
  Val x832 = (x830 - (x831 * arg0.bits[20]._super._super));
  Val x833 = (arg0.bits[214]._super._super + arg0.bits[21]._super._super);
  Val x834 = (arg0.bits[214]._super._super * Val(2));
  Val x835 = (x833 - (x834 * arg0.bits[21]._super._super));
  Val x836 = (arg0.bits[215]._super._super + arg0.bits[22]._super._super);
  Val x837 = (arg0.bits[215]._super._super * Val(2));
  Val x838 = (x836 - (x837 * arg0.bits[22]._super._super));
  Val x839 = (arg0.bits[217]._super._super + arg0.bits[24]._super._super);
  Val x840 = (arg0.bits[217]._super._super * Val(2));
  Val x841 = (x839 - (x840 * arg0.bits[24]._super._super));
  Val x842 = (arg0.bits[218]._super._super + arg0.bits[25]._super._super);
  Val x843 = (arg0.bits[218]._super._super * Val(2));
  Val x844 = (x842 - (x843 * arg0.bits[25]._super._super));
  Val x845 = (arg0.bits[219]._super._super + arg0.bits[26]._super._super);
  Val x846 = (arg0.bits[219]._super._super * Val(2));
  Val x847 = (x845 - (x846 * arg0.bits[26]._super._super));
  Val x848 = (arg0.bits[220]._super._super + arg0.bits[27]._super._super);
  Val x849 = (arg0.bits[220]._super._super * Val(2));
  Val x850 = (x848 - (x849 * arg0.bits[27]._super._super));
  Val x851 = (arg0.bits[221]._super._super + arg0.bits[28]._super._super);
  Val x852 = (arg0.bits[221]._super._super * Val(2));
  Val x853 = (x851 - (x852 * arg0.bits[28]._super._super));
  Val x854 = (arg0.bits[222]._super._super + arg0.bits[29]._super._super);
  Val x855 = (arg0.bits[222]._super._super * Val(2));
  Val x856 = (x854 - (x855 * arg0.bits[29]._super._super));
  Val x857 = (arg0.bits[223]._super._super + arg0.bits[30]._super._super);
  Val x858 = (arg0.bits[223]._super._super * Val(2));
  Val x859 = (x857 - (x858 * arg0.bits[30]._super._super));
  Val x860 = (arg0.bits[224]._super._super + arg0.bits[31]._super._super);
  Val x861 = (arg0.bits[224]._super._super * Val(2));
  Val x862 = (x860 - (x861 * arg0.bits[31]._super._super));
  Val x863 = (arg0.bits[225]._super._super + arg0.bits[32]._super._super);
  Val x864 = (arg0.bits[225]._super._super * Val(2));
  Val x865 = (x863 - (x864 * arg0.bits[32]._super._super));
  Val x866 = (arg0.bits[226]._super._super + arg0.bits[33]._super._super);
  Val x867 = (arg0.bits[226]._super._super * Val(2));
  Val x868 = (x866 - (x867 * arg0.bits[33]._super._super));
  Val x869 = (arg0.bits[227]._super._super + arg0.bits[34]._super._super);
  Val x870 = (arg0.bits[227]._super._super * Val(2));
  Val x871 = (x869 - (x870 * arg0.bits[34]._super._super));
  Val x872 = (arg0.bits[228]._super._super + arg0.bits[35]._super._super);
  Val x873 = (arg0.bits[228]._super._super * Val(2));
  Val x874 = (x872 - (x873 * arg0.bits[35]._super._super));
  Val x875 = (arg0.bits[229]._super._super + arg0.bits[36]._super._super);
  Val x876 = (arg0.bits[229]._super._super * Val(2));
  Val x877 = (x875 - (x876 * arg0.bits[36]._super._super));
  Val x878 = (arg0.bits[230]._super._super + arg0.bits[37]._super._super);
  Val x879 = (arg0.bits[230]._super._super * Val(2));
  Val x880 = (x878 - (x879 * arg0.bits[37]._super._super));
  Val x881 = (arg0.bits[231]._super._super + arg0.bits[38]._super._super);
  Val x882 = (arg0.bits[231]._super._super * Val(2));
  Val x883 = (x881 - (x882 * arg0.bits[38]._super._super));
  Val x884 = (arg0.bits[232]._super._super + arg0.bits[39]._super._super);
  Val x885 = (arg0.bits[232]._super._super * Val(2));
  Val x886 = (x884 - (x885 * arg0.bits[39]._super._super));
  Val x887 = (arg0.bits[233]._super._super + arg0.bits[40]._super._super);
  Val x888 = (arg0.bits[233]._super._super * Val(2));
  Val x889 = (x887 - (x888 * arg0.bits[40]._super._super));
  Val x890 = (arg0.bits[234]._super._super + arg0.bits[41]._super._super);
  Val x891 = (arg0.bits[234]._super._super * Val(2));
  Val x892 = (x890 - (x891 * arg0.bits[41]._super._super));
  Val x893 = (arg0.bits[235]._super._super + arg0.bits[42]._super._super);
  Val x894 = (arg0.bits[235]._super._super * Val(2));
  Val x895 = (x893 - (x894 * arg0.bits[42]._super._super));
  Val x896 = (arg0.bits[236]._super._super + arg0.bits[43]._super._super);
  Val x897 = (arg0.bits[236]._super._super * Val(2));
  Val x898 = (x896 - (x897 * arg0.bits[43]._super._super));
  Val x899 = (arg0.bits[237]._super._super + arg0.bits[44]._super._super);
  Val x900 = (arg0.bits[237]._super._super * Val(2));
  Val x901 = (x899 - (x900 * arg0.bits[44]._super._super));
  Val x902 = (arg0.bits[238]._super._super + arg0.bits[45]._super._super);
  Val x903 = (arg0.bits[238]._super._super * Val(2));
  Val x904 = (x902 - (x903 * arg0.bits[45]._super._super));
  Val x905 = (arg0.bits[239]._super._super + arg0.bits[46]._super._super);
  Val x906 = (arg0.bits[239]._super._super * Val(2));
  Val x907 = (x905 - (x906 * arg0.bits[46]._super._super));
  Val x908 = (arg0.bits[240]._super._super + arg0.bits[47]._super._super);
  Val x909 = (arg0.bits[240]._super._super * Val(2));
  Val x910 = (x908 - (x909 * arg0.bits[47]._super._super));
  Val x911 = (arg0.bits[241]._super._super + arg0.bits[48]._super._super);
  Val x912 = (arg0.bits[241]._super._super * Val(2));
  Val x913 = (x911 - (x912 * arg0.bits[48]._super._super));
  Val x914 = (arg0.bits[242]._super._super + arg0.bits[49]._super._super);
  Val x915 = (arg0.bits[242]._super._super * Val(2));
  Val x916 = (x914 - (x915 * arg0.bits[49]._super._super));
  Val x917 = (arg0.bits[243]._super._super + arg0.bits[50]._super._super);
  Val x918 = (arg0.bits[243]._super._super * Val(2));
  Val x919 = (x917 - (x918 * arg0.bits[50]._super._super));
  Val x920 = (arg0.bits[244]._super._super + arg0.bits[51]._super._super);
  Val x921 = (arg0.bits[244]._super._super * Val(2));
  Val x922 = (x920 - (x921 * arg0.bits[51]._super._super));
  Val x923 = (arg0.bits[245]._super._super + arg0.bits[52]._super._super);
  Val x924 = (arg0.bits[245]._super._super * Val(2));
  Val x925 = (x923 - (x924 * arg0.bits[52]._super._super));
  Val x926 = (arg0.bits[246]._super._super + arg0.bits[53]._super._super);
  Val x927 = (arg0.bits[246]._super._super * Val(2));
  Val x928 = (x926 - (x927 * arg0.bits[53]._super._super));
  Val x929 = (arg0.bits[247]._super._super + arg0.bits[54]._super._super);
  Val x930 = (arg0.bits[247]._super._super * Val(2));
  Val x931 = (x929 - (x930 * arg0.bits[54]._super._super));
  Val x932 = (arg0.bits[248]._super._super + arg0.bits[55]._super._super);
  Val x933 = (arg0.bits[248]._super._super * Val(2));
  Val x934 = (x932 - (x933 * arg0.bits[55]._super._super));
  Val x935 = (arg0.bits[249]._super._super + arg0.bits[56]._super._super);
  Val x936 = (arg0.bits[249]._super._super * Val(2));
  Val x937 = (x935 - (x936 * arg0.bits[56]._super._super));
  Val x938 = (arg0.bits[250]._super._super + arg0.bits[57]._super._super);
  Val x939 = (arg0.bits[250]._super._super * Val(2));
  Val x940 = (x938 - (x939 * arg0.bits[57]._super._super));
  Val x941 = (arg0.bits[251]._super._super + arg0.bits[58]._super._super);
  Val x942 = (arg0.bits[251]._super._super * Val(2));
  Val x943 = (x941 - (x942 * arg0.bits[58]._super._super));
  Val x944 = (arg0.bits[252]._super._super + arg0.bits[59]._super._super);
  Val x945 = (arg0.bits[252]._super._super * Val(2));
  Val x946 = (x944 - (x945 * arg0.bits[59]._super._super));
  Val x947 = (arg0.bits[253]._super._super + arg0.bits[60]._super._super);
  Val x948 = (arg0.bits[253]._super._super * Val(2));
  Val x949 = (x947 - (x948 * arg0.bits[60]._super._super));
  Val x950 = (arg0.bits[254]._super._super + arg0.bits[61]._super._super);
  Val x951 = (arg0.bits[254]._super._super * Val(2));
  Val x952 = (x950 - (x951 * arg0.bits[61]._super._super));
  Val x953 = (arg0.bits[255]._super._super + arg0.bits[62]._super._super);
  Val x954 = (arg0.bits[255]._super._super * Val(2));
  Val x955 = (x953 - (x954 * arg0.bits[62]._super._super));
  // ThetaP2(zirgen/circuit/keccak2/keccak.zir:31)
  Val x956 = (arg1.bits[0]._super._super + x7);
  Val x957 = (arg1.bits[0]._super._super * Val(2));
  Val x958 = (arg1.bits[1]._super._super + x10);
  Val x959 = (arg1.bits[1]._super._super * Val(2));
  Val x960 = (arg1.bits[2]._super._super + x13);
  Val x961 = (arg1.bits[2]._super._super * Val(2));
  Val x962 = (arg1.bits[3]._super._super + x16);
  Val x963 = (arg1.bits[3]._super._super * Val(2));
  Val x964 = (arg1.bits[4]._super._super + x19);
  Val x965 = (arg1.bits[4]._super._super * Val(2));
  Val x966 = (arg1.bits[5]._super._super + x22);
  Val x967 = (arg1.bits[5]._super._super * Val(2));
  Val x968 = (arg1.bits[6]._super._super + x25);
  Val x969 = (arg1.bits[6]._super._super * Val(2));
  Val x970 = (arg1.bits[7]._super._super + x28);
  Val x971 = (arg1.bits[7]._super._super * Val(2));
  Val x972 = (arg1.bits[8]._super._super + x31);
  Val x973 = (arg1.bits[8]._super._super * Val(2));
  Val x974 = (arg1.bits[9]._super._super + x34);
  Val x975 = (arg1.bits[9]._super._super * Val(2));
  Val x976 = (arg1.bits[10]._super._super + x37);
  Val x977 = (arg1.bits[10]._super._super * Val(2));
  Val x978 = (arg1.bits[11]._super._super + x40);
  Val x979 = (arg1.bits[11]._super._super * Val(2));
  Val x980 = (arg1.bits[12]._super._super + x43);
  Val x981 = (arg1.bits[12]._super._super * Val(2));
  Val x982 = (arg1.bits[13]._super._super + x46);
  Val x983 = (arg1.bits[13]._super._super * Val(2));
  Val x984 = (arg1.bits[14]._super._super + x49);
  Val x985 = (arg1.bits[14]._super._super * Val(2));
  Val x986 = (arg1.bits[15]._super._super + x52);
  Val x987 = (arg1.bits[15]._super._super * Val(2));
  Val x988 = (arg1.bits[16]._super._super + x55);
  Val x989 = (arg1.bits[16]._super._super * Val(2));
  Val x990 = (arg1.bits[17]._super._super + x58);
  Val x991 = (arg1.bits[17]._super._super * Val(2));
  Val x992 = (arg1.bits[18]._super._super + x61);
  Val x993 = (arg1.bits[18]._super._super * Val(2));
  Val x994 = (arg1.bits[19]._super._super + x64);
  Val x995 = (arg1.bits[19]._super._super * Val(2));
  Val x996 = (arg1.bits[20]._super._super + x67);
  Val x997 = (arg1.bits[20]._super._super * Val(2));
  Val x998 = (arg1.bits[21]._super._super + x70);
  Val x999 = (arg1.bits[21]._super._super * Val(2));
  Val x1000 = (arg1.bits[22]._super._super + x73);
  Val x1001 = (arg1.bits[22]._super._super * Val(2));
  Val x1002 = (arg1.bits[23]._super._super + x76);
  Val x1003 = (arg1.bits[23]._super._super * Val(2));
  Val x1004 = (arg1.bits[24]._super._super + x79);
  Val x1005 = (arg1.bits[24]._super._super * Val(2));
  Val x1006 = (arg1.bits[25]._super._super + x82);
  Val x1007 = (arg1.bits[25]._super._super * Val(2));
  Val x1008 = (arg1.bits[26]._super._super + x85);
  Val x1009 = (arg1.bits[26]._super._super * Val(2));
  Val x1010 = (arg1.bits[27]._super._super + x88);
  Val x1011 = (arg1.bits[27]._super._super * Val(2));
  Val x1012 = (arg1.bits[28]._super._super + x91);
  Val x1013 = (arg1.bits[28]._super._super * Val(2));
  Val x1014 = (arg1.bits[29]._super._super + x94);
  Val x1015 = (arg1.bits[29]._super._super * Val(2));
  Val x1016 = (arg1.bits[30]._super._super + x97);
  Val x1017 = (arg1.bits[30]._super._super * Val(2));
  Val x1018 = (arg1.bits[31]._super._super + x100);
  Val x1019 = (arg1.bits[31]._super._super * Val(2));
  Val x1020 = (arg1.bits[32]._super._super + x199);
  Val x1021 = (arg1.bits[32]._super._super * Val(2));
  Val x1022 = (arg1.bits[33]._super._super + x202);
  Val x1023 = (arg1.bits[33]._super._super * Val(2));
  Val x1024 = (arg1.bits[34]._super._super + x205);
  Val x1025 = (arg1.bits[34]._super._super * Val(2));
  Val x1026 = (arg1.bits[35]._super._super + x208);
  Val x1027 = (arg1.bits[35]._super._super * Val(2));
  Val x1028 = (arg1.bits[36]._super._super + x211);
  Val x1029 = (arg1.bits[36]._super._super * Val(2));
  Val x1030 = (arg1.bits[37]._super._super + x214);
  Val x1031 = (arg1.bits[37]._super._super * Val(2));
  Val x1032 = (arg1.bits[38]._super._super + x217);
  Val x1033 = (arg1.bits[38]._super._super * Val(2));
  Val x1034 = (arg1.bits[39]._super._super + x220);
  Val x1035 = (arg1.bits[39]._super._super * Val(2));
  Val x1036 = (arg1.bits[40]._super._super + x223);
  Val x1037 = (arg1.bits[40]._super._super * Val(2));
  Val x1038 = (arg1.bits[41]._super._super + x226);
  Val x1039 = (arg1.bits[41]._super._super * Val(2));
  Val x1040 = (arg1.bits[42]._super._super + x229);
  Val x1041 = (arg1.bits[42]._super._super * Val(2));
  Val x1042 = (arg1.bits[43]._super._super + x232);
  Val x1043 = (arg1.bits[43]._super._super * Val(2));
  Val x1044 = (arg1.bits[44]._super._super + x235);
  Val x1045 = (arg1.bits[44]._super._super * Val(2));
  Val x1046 = (arg1.bits[45]._super._super + x238);
  Val x1047 = (arg1.bits[45]._super._super * Val(2));
  Val x1048 = (arg1.bits[46]._super._super + x241);
  Val x1049 = (arg1.bits[46]._super._super * Val(2));
  Val x1050 = (arg1.bits[47]._super._super + x244);
  Val x1051 = (arg1.bits[47]._super._super * Val(2));
  Val x1052 = (arg1.bits[48]._super._super + x247);
  Val x1053 = (arg1.bits[48]._super._super * Val(2));
  Val x1054 = (arg1.bits[49]._super._super + x250);
  Val x1055 = (arg1.bits[49]._super._super * Val(2));
  Val x1056 = (arg1.bits[50]._super._super + x253);
  Val x1057 = (arg1.bits[50]._super._super * Val(2));
  Val x1058 = (arg1.bits[51]._super._super + x256);
  Val x1059 = (arg1.bits[51]._super._super * Val(2));
  Val x1060 = (arg1.bits[52]._super._super + x259);
  Val x1061 = (arg1.bits[52]._super._super * Val(2));
  Val x1062 = (arg1.bits[53]._super._super + x262);
  Val x1063 = (arg1.bits[53]._super._super * Val(2));
  Val x1064 = (arg1.bits[54]._super._super + x265);
  Val x1065 = (arg1.bits[54]._super._super * Val(2));
  Val x1066 = (arg1.bits[55]._super._super + x268);
  Val x1067 = (arg1.bits[55]._super._super * Val(2));
  Val x1068 = (arg1.bits[56]._super._super + x271);
  Val x1069 = (arg1.bits[56]._super._super * Val(2));
  Val x1070 = (arg1.bits[57]._super._super + x274);
  Val x1071 = (arg1.bits[57]._super._super * Val(2));
  Val x1072 = (arg1.bits[58]._super._super + x277);
  Val x1073 = (arg1.bits[58]._super._super * Val(2));
  Val x1074 = (arg1.bits[59]._super._super + x280);
  Val x1075 = (arg1.bits[59]._super._super * Val(2));
  Val x1076 = (arg1.bits[60]._super._super + x283);
  Val x1077 = (arg1.bits[60]._super._super * Val(2));
  Val x1078 = (arg1.bits[61]._super._super + x286);
  Val x1079 = (arg1.bits[61]._super._super * Val(2));
  Val x1080 = (arg1.bits[62]._super._super + x289);
  Val x1081 = (arg1.bits[62]._super._super * Val(2));
  Val x1082 = (arg2.bits[63]._super._super + x382);
  Val x1083 = (arg2.bits[63]._super._super * Val(2));
  Val x1084 = (arg1.bits[66]._super._super + x391);
  Val x1085 = (arg1.bits[66]._super._super * Val(2));
  Val x1086 = (arg1.bits[67]._super._super + x394);
  Val x1087 = (arg1.bits[67]._super._super * Val(2));
  Val x1088 = (arg1.bits[68]._super._super + x397);
  Val x1089 = (arg1.bits[68]._super._super * Val(2));
  Val x1090 = (arg1.bits[69]._super._super + x400);
  Val x1091 = (arg1.bits[69]._super._super * Val(2));
  Val x1092 = (arg1.bits[70]._super._super + x403);
  Val x1093 = (arg1.bits[70]._super._super * Val(2));
  Val x1094 = (arg1.bits[71]._super._super + x406);
  Val x1095 = (arg1.bits[71]._super._super * Val(2));
  Val x1096 = (arg1.bits[72]._super._super + x409);
  Val x1097 = (arg1.bits[72]._super._super * Val(2));
  Val x1098 = (arg1.bits[73]._super._super + x412);
  Val x1099 = (arg1.bits[73]._super._super * Val(2));
  Val x1100 = (arg1.bits[74]._super._super + x415);
  Val x1101 = (arg1.bits[74]._super._super * Val(2));
  Val x1102 = (arg1.bits[75]._super._super + x418);
  Val x1103 = (arg1.bits[75]._super._super * Val(2));
  Val x1104 = (arg1.bits[76]._super._super + x421);
  Val x1105 = (arg1.bits[76]._super._super * Val(2));
  Val x1106 = (arg1.bits[77]._super._super + x424);
  Val x1107 = (arg1.bits[77]._super._super * Val(2));
  Val x1108 = (arg1.bits[78]._super._super + x427);
  Val x1109 = (arg1.bits[78]._super._super * Val(2));
  Val x1110 = (arg1.bits[79]._super._super + x430);
  Val x1111 = (arg1.bits[79]._super._super * Val(2));
  Val x1112 = (arg1.bits[80]._super._super + x433);
  Val x1113 = (arg1.bits[80]._super._super * Val(2));
  Val x1114 = (arg1.bits[81]._super._super + x436);
  Val x1115 = (arg1.bits[81]._super._super * Val(2));
  Val x1116 = (arg1.bits[82]._super._super + x439);
  Val x1117 = (arg1.bits[82]._super._super * Val(2));
  Val x1118 = (arg1.bits[83]._super._super + x442);
  Val x1119 = (arg1.bits[83]._super._super * Val(2));
  Val x1120 = (arg1.bits[84]._super._super + x445);
  Val x1121 = (arg1.bits[84]._super._super * Val(2));
  Val x1122 = (arg1.bits[85]._super._super + x448);
  Val x1123 = (arg1.bits[85]._super._super * Val(2));
  Val x1124 = (arg1.bits[86]._super._super + x451);
  Val x1125 = (arg1.bits[86]._super._super * Val(2));
  Val x1126 = (arg1.bits[87]._super._super + x454);
  Val x1127 = (arg1.bits[87]._super._super * Val(2));
  Val x1128 = (arg1.bits[88]._super._super + x457);
  Val x1129 = (arg1.bits[88]._super._super * Val(2));
  Val x1130 = (arg1.bits[89]._super._super + x460);
  Val x1131 = (arg1.bits[89]._super._super * Val(2));
  Val x1132 = (arg1.bits[90]._super._super + x463);
  Val x1133 = (arg1.bits[90]._super._super * Val(2));
  Val x1134 = (arg1.bits[91]._super._super + x466);
  Val x1135 = (arg1.bits[91]._super._super * Val(2));
  Val x1136 = (arg1.bits[92]._super._super + x469);
  Val x1137 = (arg1.bits[92]._super._super * Val(2));
  Val x1138 = (arg1.bits[93]._super._super + x472);
  Val x1139 = (arg1.bits[93]._super._super * Val(2));
  Val x1140 = (arg1.bits[94]._super._super + x475);
  Val x1141 = (arg1.bits[94]._super._super * Val(2));
  Val x1142 = (arg1.bits[95]._super._super + x478);
  Val x1143 = (arg1.bits[95]._super._super * Val(2));
  Val x1144 = (arg2.bits[64]._super._super + x481);
  Val x1145 = (arg2.bits[64]._super._super * Val(2));
  Val x1146 = (arg2.bits[65]._super._super + x484);
  Val x1147 = (arg2.bits[65]._super._super * Val(2));
  Val x1148 = (arg1.bits[96]._super._super + x577);
  Val x1149 = (arg1.bits[96]._super._super * Val(2));
  Val x1150 = (arg1.bits[97]._super._super + x580);
  Val x1151 = (arg1.bits[97]._super._super * Val(2));
  Val x1152 = (arg1.bits[98]._super._super + x583);
  Val x1153 = (arg1.bits[98]._super._super * Val(2));
  Val x1154 = (arg1.bits[99]._super._super + x586);
  Val x1155 = (arg1.bits[99]._super._super * Val(2));
  Val x1156 = (arg2.bits[100]._super._super + x685);
  Val x1157 = (arg2.bits[100]._super._super * Val(2));
  Val x1158 = (arg2.bits[101]._super._super + x688);
  Val x1159 = (arg2.bits[101]._super._super * Val(2));
  Val x1160 = (arg2.bits[102]._super._super + x691);
  Val x1161 = (arg2.bits[102]._super._super * Val(2));
  Val x1162 = (arg2.bits[103]._super._super + x694);
  Val x1163 = (arg2.bits[103]._super._super * Val(2));
  Val x1164 = (arg2.bits[104]._super._super + x697);
  Val x1165 = (arg2.bits[104]._super._super * Val(2));
  Val x1166 = (arg2.bits[105]._super._super + x700);
  Val x1167 = (arg2.bits[105]._super._super * Val(2));
  Val x1168 = (arg2.bits[106]._super._super + x703);
  Val x1169 = (arg2.bits[106]._super._super * Val(2));
  Val x1170 = (arg2.bits[107]._super._super + x706);
  Val x1171 = (arg2.bits[107]._super._super * Val(2));
  Val x1172 = (arg2.bits[108]._super._super + x709);
  Val x1173 = (arg2.bits[108]._super._super * Val(2));
  Val x1174 = (arg2.bits[109]._super._super + x712);
  Val x1175 = (arg2.bits[109]._super._super * Val(2));
  Val x1176 = (arg2.bits[110]._super._super + x715);
  Val x1177 = (arg2.bits[110]._super._super * Val(2));
  Val x1178 = (arg2.bits[111]._super._super + x718);
  Val x1179 = (arg2.bits[111]._super._super * Val(2));
  Val x1180 = (arg2.bits[112]._super._super + x721);
  Val x1181 = (arg2.bits[112]._super._super * Val(2));
  Val x1182 = (arg2.bits[113]._super._super + x724);
  Val x1183 = (arg2.bits[113]._super._super * Val(2));
  Val x1184 = (arg2.bits[114]._super._super + x727);
  Val x1185 = (arg2.bits[114]._super._super * Val(2));
  Val x1186 = (arg2.bits[115]._super._super + x730);
  Val x1187 = (arg2.bits[115]._super._super * Val(2));
  Val x1188 = (arg2.bits[116]._super._super + x733);
  Val x1189 = (arg2.bits[116]._super._super * Val(2));
  Val x1190 = (arg2.bits[117]._super._super + x736);
  Val x1191 = (arg2.bits[117]._super._super * Val(2));
  Val x1192 = (arg2.bits[118]._super._super + x739);
  Val x1193 = (arg2.bits[118]._super._super * Val(2));
  Val x1194 = (arg2.bits[119]._super._super + x742);
  Val x1195 = (arg2.bits[119]._super._super * Val(2));
  Val x1196 = (arg2.bits[120]._super._super + x745);
  Val x1197 = (arg2.bits[120]._super._super * Val(2));
  Val x1198 = (arg2.bits[121]._super._super + x748);
  Val x1199 = (arg2.bits[121]._super._super * Val(2));
  Val x1200 = (arg2.bits[122]._super._super + x751);
  Val x1201 = (arg2.bits[122]._super._super * Val(2));
  Val x1202 = (arg2.bits[123]._super._super + x754);
  Val x1203 = (arg2.bits[123]._super._super * Val(2));
  Val x1204 = (arg2.bits[124]._super._super + x757);
  Val x1205 = (arg2.bits[124]._super._super * Val(2));
  Val x1206 = (arg2.bits[125]._super._super + x760);
  Val x1207 = (arg2.bits[125]._super._super * Val(2));
  Val x1208 = (arg2.bits[126]._super._super + x763);
  Val x1209 = (arg2.bits[126]._super._super * Val(2));
  Val x1210 = (arg2.bits[127]._super._super + x766);
  Val x1211 = (arg2.bits[127]._super._super * Val(2));
  Val x1212 = (arg1.bits[128]._super._super + x769);
  Val x1213 = (arg1.bits[128]._super._super * Val(2));
  Val x1214 = (arg1.bits[129]._super._super + x772);
  Val x1215 = (arg1.bits[129]._super._super * Val(2));
  Val x1216 = (arg1.bits[130]._super._super + x775);
  Val x1217 = (arg1.bits[130]._super._super * Val(2));
  Val x1218 = (arg1.bits[131]._super._super + x778);
  Val x1219 = (arg1.bits[131]._super._super * Val(2));
  Val x1220 = (arg1.bits[132]._super._super + x781);
  Val x1221 = (arg1.bits[132]._super._super * Val(2));
  Val x1222 = (arg2.bits[133]._super._super + x877);
  Val x1223 = (arg2.bits[133]._super._super * Val(2));
  Val x1224 = (arg2.bits[134]._super._super + x880);
  Val x1225 = (arg2.bits[134]._super._super * Val(2));
  Val x1226 = (arg2.bits[135]._super._super + x883);
  Val x1227 = (arg2.bits[135]._super._super * Val(2));
  Val x1228 = (arg2.bits[136]._super._super + x886);
  Val x1229 = (arg2.bits[136]._super._super * Val(2));
  Val x1230 = (arg2.bits[137]._super._super + x889);
  Val x1231 = (arg2.bits[137]._super._super * Val(2));
  Val x1232 = (arg2.bits[138]._super._super + x892);
  Val x1233 = (arg2.bits[138]._super._super * Val(2));
  Val x1234 = (arg2.bits[139]._super._super + x895);
  Val x1235 = (arg2.bits[139]._super._super * Val(2));
  Val x1236 = (arg2.bits[140]._super._super + x898);
  Val x1237 = (arg2.bits[140]._super._super * Val(2));
  Val x1238 = (arg2.bits[141]._super._super + x901);
  Val x1239 = (arg2.bits[141]._super._super * Val(2));
  Val x1240 = (arg2.bits[142]._super._super + x904);
  Val x1241 = (arg2.bits[142]._super._super * Val(2));
  Val x1242 = (arg2.bits[143]._super._super + x907);
  Val x1243 = (arg2.bits[143]._super._super * Val(2));
  Val x1244 = (arg2.bits[144]._super._super + x910);
  Val x1245 = (arg2.bits[144]._super._super * Val(2));
  Val x1246 = (arg2.bits[145]._super._super + x913);
  Val x1247 = (arg2.bits[145]._super._super * Val(2));
  Val x1248 = (arg2.bits[146]._super._super + x916);
  Val x1249 = (arg2.bits[146]._super._super * Val(2));
  Val x1250 = (arg2.bits[147]._super._super + x919);
  Val x1251 = (arg2.bits[147]._super._super * Val(2));
  Val x1252 = (arg2.bits[148]._super._super + x922);
  Val x1253 = (arg2.bits[148]._super._super * Val(2));
  Val x1254 = (arg2.bits[149]._super._super + x925);
  Val x1255 = (arg2.bits[149]._super._super * Val(2));
  Val x1256 = (arg2.bits[150]._super._super + x928);
  Val x1257 = (arg2.bits[150]._super._super * Val(2));
  Val x1258 = (arg2.bits[151]._super._super + x931);
  Val x1259 = (arg2.bits[151]._super._super * Val(2));
  Val x1260 = (arg2.bits[152]._super._super + x934);
  Val x1261 = (arg2.bits[152]._super._super * Val(2));
  Val x1262 = (arg2.bits[153]._super._super + x937);
  Val x1263 = (arg2.bits[153]._super._super * Val(2));
  Val x1264 = (arg2.bits[154]._super._super + x940);
  Val x1265 = (arg2.bits[154]._super._super * Val(2));
  Val x1266 = (arg2.bits[155]._super._super + x943);
  Val x1267 = (arg2.bits[155]._super._super * Val(2));
  Val x1268 = (arg2.bits[156]._super._super + x946);
  Val x1269 = (arg2.bits[156]._super._super * Val(2));
  Val x1270 = (arg2.bits[157]._super._super + x949);
  Val x1271 = (arg2.bits[157]._super._super * Val(2));
  Val x1272 = (arg2.bits[158]._super._super + x952);
  Val x1273 = (arg2.bits[158]._super._super * Val(2));
  Val x1274 = (arg2.bits[159]._super._super + x955);
  Val x1275 = (arg2.bits[159]._super._super * Val(2));
  Val x1276 = (arg1.bits[188]._super._super + x91);
  Val x1277 = (arg1.bits[188]._super._super * Val(2));
  Val x1278 = (arg1.bits[189]._super._super + x94);
  Val x1279 = (arg1.bits[189]._super._super * Val(2));
  Val x1280 = (arg1.bits[190]._super._super + x97);
  Val x1281 = (arg1.bits[190]._super._super * Val(2));
  Val x1282 = (arg1.bits[191]._super._super + x100);
  Val x1283 = (arg1.bits[191]._super._super * Val(2));
  Val x1284 = (arg2.bits[160]._super._super + x103);
  Val x1285 = (arg2.bits[160]._super._super * Val(2));
  Val x1286 = (arg2.bits[161]._super._super + x106);
  Val x1287 = (arg2.bits[161]._super._super * Val(2));
  Val x1288 = (arg2.bits[162]._super._super + x109);
  Val x1289 = (arg2.bits[162]._super._super * Val(2));
  Val x1290 = (arg2.bits[163]._super._super + x112);
  Val x1291 = (arg2.bits[163]._super._super * Val(2));
  Val x1292 = (arg2.bits[164]._super._super + x115);
  Val x1293 = (arg2.bits[164]._super._super * Val(2));
  Val x1294 = (arg2.bits[165]._super._super + x118);
  Val x1295 = (arg2.bits[165]._super._super * Val(2));
  Val x1296 = (arg2.bits[166]._super._super + x121);
  Val x1297 = (arg2.bits[166]._super._super * Val(2));
  Val x1298 = (arg2.bits[167]._super._super + x124);
  Val x1299 = (arg2.bits[167]._super._super * Val(2));
  Val x1300 = (arg2.bits[168]._super._super + x127);
  Val x1301 = (arg2.bits[168]._super._super * Val(2));
  Val x1302 = (arg2.bits[169]._super._super + x130);
  Val x1303 = (arg2.bits[169]._super._super * Val(2));
  Val x1304 = (arg2.bits[170]._super._super + x133);
  Val x1305 = (arg2.bits[170]._super._super * Val(2));
  Val x1306 = (arg2.bits[171]._super._super + x136);
  Val x1307 = (arg2.bits[171]._super._super * Val(2));
  Val x1308 = (arg2.bits[172]._super._super + x139);
  Val x1309 = (arg2.bits[172]._super._super * Val(2));
  Val x1310 = (arg2.bits[173]._super._super + x142);
  Val x1311 = (arg2.bits[173]._super._super * Val(2));
  Val x1312 = (arg2.bits[174]._super._super + x145);
  Val x1313 = (arg2.bits[174]._super._super * Val(2));
  Val x1314 = (arg2.bits[175]._super._super + x148);
  Val x1315 = (arg2.bits[175]._super._super * Val(2));
  Val x1316 = (arg2.bits[176]._super._super + x151);
  Val x1317 = (arg2.bits[176]._super._super * Val(2));
  Val x1318 = (arg2.bits[177]._super._super + x154);
  Val x1319 = (arg2.bits[177]._super._super * Val(2));
  Val x1320 = (arg2.bits[178]._super._super + x157);
  Val x1321 = (arg2.bits[178]._super._super * Val(2));
  Val x1322 = (arg2.bits[179]._super._super + x160);
  Val x1323 = (arg2.bits[179]._super._super * Val(2));
  Val x1324 = (arg2.bits[180]._super._super + x163);
  Val x1325 = (arg2.bits[180]._super._super * Val(2));
  Val x1326 = (arg2.bits[181]._super._super + x166);
  Val x1327 = (arg2.bits[181]._super._super * Val(2));
  Val x1328 = (arg2.bits[182]._super._super + x169);
  Val x1329 = (arg2.bits[182]._super._super * Val(2));
  Val x1330 = (arg2.bits[183]._super._super + x172);
  Val x1331 = (arg2.bits[183]._super._super * Val(2));
  Val x1332 = (arg2.bits[184]._super._super + x175);
  Val x1333 = (arg2.bits[184]._super._super * Val(2));
  Val x1334 = (arg2.bits[185]._super._super + x178);
  Val x1335 = (arg2.bits[185]._super._super * Val(2));
  Val x1336 = (arg2.bits[186]._super._super + x181);
  Val x1337 = (arg2.bits[186]._super._super * Val(2));
  Val x1338 = (arg2.bits[187]._super._super + x184);
  Val x1339 = (arg2.bits[187]._super._super * Val(2));
  Val x1340 = (arg1.bits[212]._super._super + x259);
  Val x1341 = (arg1.bits[212]._super._super * Val(2));
  Val x1342 = (arg1.bits[213]._super._super + x262);
  Val x1343 = (arg1.bits[213]._super._super * Val(2));
  Val x1344 = (arg1.bits[214]._super._super + x265);
  Val x1345 = (arg1.bits[214]._super._super * Val(2));
  Val x1346 = (arg1.bits[215]._super._super + x268);
  Val x1347 = (arg1.bits[215]._super._super * Val(2));
  Val x1348 = (arg1.bits[216]._super._super + x271);
  Val x1349 = (arg1.bits[216]._super._super * Val(2));
  Val x1350 = (arg1.bits[217]._super._super + x274);
  Val x1351 = (arg1.bits[217]._super._super * Val(2));
  Val x1352 = (arg1.bits[218]._super._super + x277);
  Val x1353 = (arg1.bits[218]._super._super * Val(2));
  Val x1354 = (arg1.bits[219]._super._super + x280);
  Val x1355 = (arg1.bits[219]._super._super * Val(2));
  Val x1356 = (arg1.bits[220]._super._super + x283);
  Val x1357 = (arg1.bits[220]._super._super * Val(2));
  Val x1358 = (arg1.bits[221]._super._super + x286);
  Val x1359 = (arg1.bits[221]._super._super * Val(2));
  Val x1360 = (arg1.bits[222]._super._super + x289);
  Val x1361 = (arg1.bits[222]._super._super * Val(2));
  Val x1362 = (arg1.bits[223]._super._super + x292);
  Val x1363 = (arg1.bits[223]._super._super * Val(2));
  Val x1364 = (arg2.bits[192]._super._super + x295);
  Val x1365 = (arg2.bits[192]._super._super * Val(2));
  Val x1366 = (arg2.bits[193]._super._super + x298);
  Val x1367 = (arg2.bits[193]._super._super * Val(2));
  Val x1368 = (arg2.bits[194]._super._super + x301);
  Val x1369 = (arg2.bits[194]._super._super * Val(2));
  Val x1370 = (arg2.bits[195]._super._super + x304);
  Val x1371 = (arg2.bits[195]._super._super * Val(2));
  Val x1372 = (arg2.bits[196]._super._super + x307);
  Val x1373 = (arg2.bits[196]._super._super * Val(2));
  Val x1374 = (arg2.bits[197]._super._super + x310);
  Val x1375 = (arg2.bits[197]._super._super * Val(2));
  Val x1376 = (arg2.bits[198]._super._super + x313);
  Val x1377 = (arg2.bits[198]._super._super * Val(2));
  Val x1378 = (arg2.bits[199]._super._super + x316);
  Val x1379 = (arg2.bits[199]._super._super * Val(2));
  Val x1380 = (arg2.bits[200]._super._super + x319);
  Val x1381 = (arg2.bits[200]._super._super * Val(2));
  Val x1382 = (arg2.bits[201]._super._super + x322);
  Val x1383 = (arg2.bits[201]._super._super * Val(2));
  Val x1384 = (arg2.bits[202]._super._super + x325);
  Val x1385 = (arg2.bits[202]._super._super * Val(2));
  Val x1386 = (arg2.bits[203]._super._super + x328);
  Val x1387 = (arg2.bits[203]._super._super * Val(2));
  Val x1388 = (arg2.bits[204]._super._super + x331);
  Val x1389 = (arg2.bits[204]._super._super * Val(2));
  Val x1390 = (arg2.bits[205]._super._super + x334);
  Val x1391 = (arg2.bits[205]._super._super * Val(2));
  Val x1392 = (arg2.bits[206]._super._super + x337);
  Val x1393 = (arg2.bits[206]._super._super * Val(2));
  Val x1394 = (arg2.bits[207]._super._super + x340);
  Val x1395 = (arg2.bits[207]._super._super * Val(2));
  Val x1396 = (arg2.bits[208]._super._super + x343);
  Val x1397 = (arg2.bits[208]._super._super * Val(2));
  Val x1398 = (arg2.bits[209]._super._super + x346);
  Val x1399 = (arg2.bits[209]._super._super * Val(2));
  Val x1400 = (arg2.bits[210]._super._super + x349);
  Val x1401 = (arg2.bits[210]._super._super * Val(2));
  Val x1402 = (arg2.bits[211]._super._super + x352);
  Val x1403 = (arg2.bits[211]._super._super * Val(2));
  Val x1404 = (arg1.bits[224]._super._super + x385);
  Val x1405 = (arg1.bits[224]._super._super * Val(2));
  Val x1406 = (arg1.bits[225]._super._super + x388);
  Val x1407 = (arg1.bits[225]._super._super * Val(2));
  Val x1408 = (arg1.bits[226]._super._super + x391);
  Val x1409 = (arg1.bits[226]._super._super * Val(2));
  Val x1410 = (arg1.bits[227]._super._super + x394);
  Val x1411 = (arg1.bits[227]._super._super * Val(2));
  Val x1412 = (arg1.bits[228]._super._super + x397);
  Val x1413 = (arg1.bits[228]._super._super * Val(2));
  Val x1414 = (arg1.bits[229]._super._super + x400);
  Val x1415 = (arg1.bits[229]._super._super * Val(2));
  Val x1416 = (arg1.bits[230]._super._super + x403);
  Val x1417 = (arg1.bits[230]._super._super * Val(2));
  Val x1418 = (arg1.bits[231]._super._super + x406);
  Val x1419 = (arg1.bits[231]._super._super * Val(2));
  Val x1420 = (arg1.bits[232]._super._super + x409);
  Val x1421 = (arg1.bits[232]._super._super * Val(2));
  Val x1422 = (arg1.bits[233]._super._super + x412);
  Val x1423 = (arg1.bits[233]._super._super * Val(2));
  Val x1424 = (arg1.bits[234]._super._super + x415);
  Val x1425 = (arg1.bits[234]._super._super * Val(2));
  Val x1426 = (arg1.bits[235]._super._super + x418);
  Val x1427 = (arg1.bits[235]._super._super * Val(2));
  Val x1428 = (arg1.bits[236]._super._super + x421);
  Val x1429 = (arg1.bits[236]._super._super * Val(2));
  Val x1430 = (arg1.bits[237]._super._super + x424);
  Val x1431 = (arg1.bits[237]._super._super * Val(2));
  Val x1432 = (arg1.bits[238]._super._super + x427);
  Val x1433 = (arg1.bits[238]._super._super * Val(2));
  Val x1434 = (arg1.bits[239]._super._super + x430);
  Val x1435 = (arg1.bits[239]._super._super * Val(2));
  Val x1436 = (arg1.bits[240]._super._super + x433);
  Val x1437 = (arg1.bits[240]._super._super * Val(2));
  Val x1438 = (arg1.bits[241]._super._super + x436);
  Val x1439 = (arg1.bits[241]._super._super * Val(2));
  Val x1440 = (arg1.bits[242]._super._super + x439);
  Val x1441 = (arg1.bits[242]._super._super * Val(2));
  Val x1442 = (arg1.bits[243]._super._super + x442);
  Val x1443 = (arg1.bits[243]._super._super * Val(2));
  Val x1444 = (arg1.bits[244]._super._super + x445);
  Val x1445 = (arg1.bits[244]._super._super * Val(2));
  Val x1446 = (arg1.bits[245]._super._super + x448);
  Val x1447 = (arg1.bits[245]._super._super * Val(2));
  Val x1448 = (arg1.bits[246]._super._super + x451);
  Val x1449 = (arg1.bits[246]._super._super * Val(2));
  Val x1450 = (arg1.bits[247]._super._super + x454);
  Val x1451 = (arg1.bits[247]._super._super * Val(2));
  Val x1452 = (arg1.bits[248]._super._super + x457);
  Val x1453 = (arg1.bits[248]._super._super * Val(2));
  Val x1454 = (arg1.bits[249]._super._super + x460);
  Val x1455 = (arg1.bits[249]._super._super * Val(2));
  Val x1456 = (arg2.bits[250]._super._super + x559);
  Val x1457 = (arg2.bits[250]._super._super * Val(2));
  Val x1458 = (arg2.bits[251]._super._super + x562);
  Val x1459 = (arg2.bits[251]._super._super * Val(2));
  Val x1460 = (arg2.bits[252]._super._super + x565);
  Val x1461 = (arg2.bits[252]._super._super * Val(2));
  Val x1462 = (arg2.bits[253]._super._super + x568);
  Val x1463 = (arg2.bits[253]._super._super * Val(2));
  Val x1464 = (arg2.bits[254]._super._super + x571);
  Val x1465 = (arg2.bits[254]._super._super * Val(2));
  Val x1466 = (arg2.bits[255]._super._super + x574);
  Val x1467 = (arg2.bits[255]._super._super * Val(2));
  Val x1468 = (arg1.bits[265]._super._super + x604);
  Val x1469 = (arg1.bits[265]._super._super * Val(2));
  Val x1470 = (arg1.bits[266]._super._super + x607);
  Val x1471 = (arg1.bits[266]._super._super * Val(2));
  Val x1472 = (arg1.bits[267]._super._super + x610);
  Val x1473 = (arg1.bits[267]._super._super * Val(2));
  Val x1474 = (arg1.bits[268]._super._super + x613);
  Val x1475 = (arg1.bits[268]._super._super * Val(2));
  Val x1476 = (arg1.bits[269]._super._super + x616);
  Val x1477 = (arg1.bits[269]._super._super * Val(2));
  Val x1478 = (arg1.bits[270]._super._super + x619);
  Val x1479 = (arg1.bits[270]._super._super * Val(2));
  Val x1480 = (arg1.bits[271]._super._super + x622);
  Val x1481 = (arg1.bits[271]._super._super * Val(2));
  Val x1482 = (arg1.bits[272]._super._super + x625);
  Val x1483 = (arg1.bits[272]._super._super * Val(2));
  Val x1484 = (arg1.bits[273]._super._super + x628);
  Val x1485 = (arg1.bits[273]._super._super * Val(2));
  Val x1486 = (arg1.bits[274]._super._super + x631);
  Val x1487 = (arg1.bits[274]._super._super * Val(2));
  Val x1488 = (arg1.bits[275]._super._super + x634);
  Val x1489 = (arg1.bits[275]._super._super * Val(2));
  Val x1490 = (arg1.bits[276]._super._super + x637);
  Val x1491 = (arg1.bits[276]._super._super * Val(2));
  Val x1492 = (arg1.bits[277]._super._super + x640);
  Val x1493 = (arg1.bits[277]._super._super * Val(2));
  Val x1494 = (arg1.bits[278]._super._super + x643);
  Val x1495 = (arg1.bits[278]._super._super * Val(2));
  Val x1496 = (arg1.bits[279]._super._super + x646);
  Val x1497 = (arg1.bits[279]._super._super * Val(2));
  Val x1498 = (arg1.bits[280]._super._super + x649);
  Val x1499 = (arg1.bits[280]._super._super * Val(2));
  Val x1500 = (arg1.bits[281]._super._super + x652);
  Val x1501 = (arg1.bits[281]._super._super * Val(2));
  Val x1502 = (arg1.bits[282]._super._super + x655);
  Val x1503 = (arg1.bits[282]._super._super * Val(2));
  Val x1504 = (arg1.bits[283]._super._super + x658);
  Val x1505 = (arg1.bits[283]._super._super * Val(2));
  Val x1506 = (arg1.bits[284]._super._super + x661);
  Val x1507 = (arg1.bits[284]._super._super * Val(2));
  Val x1508 = (arg1.bits[285]._super._super + x664);
  Val x1509 = (arg1.bits[285]._super._super * Val(2));
  Val x1510 = (arg1.bits[286]._super._super + x667);
  Val x1511 = (arg1.bits[286]._super._super * Val(2));
  Val x1512 = (arg1.bits[287]._super._super + x670);
  Val x1513 = (arg1.bits[287]._super._super * Val(2));
  Val x1514 = (arg2.bits[256]._super._super + x673);
  Val x1515 = (arg2.bits[256]._super._super * Val(2));
  Val x1516 = (arg2.bits[257]._super._super + x676);
  Val x1517 = (arg2.bits[257]._super._super * Val(2));
  Val x1518 = (arg2.bits[258]._super._super + x679);
  Val x1519 = (arg2.bits[258]._super._super * Val(2));
  Val x1520 = (arg2.bits[259]._super._super + x682);
  Val x1521 = (arg2.bits[259]._super._super * Val(2));
  Val x1522 = (arg2.bits[260]._super._super + x685);
  Val x1523 = (arg2.bits[260]._super._super * Val(2));
  Val x1524 = (arg2.bits[261]._super._super + x688);
  Val x1525 = (arg2.bits[261]._super._super * Val(2));
  Val x1526 = (arg2.bits[262]._super._super + x691);
  Val x1527 = (arg2.bits[262]._super._super * Val(2));
  Val x1528 = (arg2.bits[263]._super._super + x694);
  Val x1529 = (arg2.bits[263]._super._super * Val(2));
  Val x1530 = (arg2.bits[264]._super._super + x697);
  Val x1531 = (arg2.bits[264]._super._super * Val(2));
  Val x1532 = (arg1.bits[288]._super._super + x769);
  Val x1533 = (arg1.bits[288]._super._super * Val(2));
  Val x1534 = (arg1.bits[289]._super._super + x772);
  Val x1535 = (arg1.bits[289]._super._super * Val(2));
  Val x1536 = (arg1.bits[290]._super._super + x775);
  Val x1537 = (arg1.bits[290]._super._super * Val(2));
  Val x1538 = (arg1.bits[291]._super._super + x778);
  Val x1539 = (arg1.bits[291]._super._super * Val(2));
  Val x1540 = (arg1.bits[292]._super._super + x781);
  Val x1541 = (arg1.bits[292]._super._super * Val(2));
  Val x1542 = (arg1.bits[293]._super._super + x784);
  Val x1543 = (arg1.bits[293]._super._super * Val(2));
  Val x1544 = (arg1.bits[294]._super._super + x787);
  Val x1545 = (arg1.bits[294]._super._super * Val(2));
  Val x1546 = (arg1.bits[295]._super._super + x790);
  Val x1547 = (arg1.bits[295]._super._super * Val(2));
  Val x1548 = (arg1.bits[296]._super._super + x793);
  Val x1549 = (arg1.bits[296]._super._super * Val(2));
  Val x1550 = (arg1.bits[297]._super._super + x796);
  Val x1551 = (arg1.bits[297]._super._super * Val(2));
  Val x1552 = (arg1.bits[298]._super._super + x799);
  Val x1553 = (arg1.bits[298]._super._super * Val(2));
  Val x1554 = (arg1.bits[299]._super._super + x802);
  Val x1555 = (arg1.bits[299]._super._super * Val(2));
  Val x1556 = (arg2.bits[300]._super._super + x898);
  Val x1557 = (arg2.bits[300]._super._super * Val(2));
  Val x1558 = (arg2.bits[301]._super._super + x901);
  Val x1559 = (arg2.bits[301]._super._super * Val(2));
  Val x1560 = (arg2.bits[302]._super._super + x904);
  Val x1561 = (arg2.bits[302]._super._super * Val(2));
  Val x1562 = (arg2.bits[303]._super._super + x907);
  Val x1563 = (arg2.bits[303]._super._super * Val(2));
  Val x1564 = (arg2.bits[304]._super._super + x910);
  Val x1565 = (arg2.bits[304]._super._super * Val(2));
  Val x1566 = (arg2.bits[305]._super._super + x913);
  Val x1567 = (arg2.bits[305]._super._super * Val(2));
  Val x1568 = (arg2.bits[306]._super._super + x916);
  Val x1569 = (arg2.bits[306]._super._super * Val(2));
  Val x1570 = (arg2.bits[307]._super._super + x919);
  Val x1571 = (arg2.bits[307]._super._super * Val(2));
  Val x1572 = (arg2.bits[308]._super._super + x922);
  Val x1573 = (arg2.bits[308]._super._super * Val(2));
  Val x1574 = (arg2.bits[309]._super._super + x925);
  Val x1575 = (arg2.bits[309]._super._super * Val(2));
  Val x1576 = (arg2.bits[310]._super._super + x928);
  Val x1577 = (arg2.bits[310]._super._super * Val(2));
  Val x1578 = (arg2.bits[311]._super._super + x931);
  Val x1579 = (arg2.bits[311]._super._super * Val(2));
  Val x1580 = (arg2.bits[312]._super._super + x934);
  Val x1581 = (arg2.bits[312]._super._super * Val(2));
  Val x1582 = (arg2.bits[313]._super._super + x937);
  Val x1583 = (arg2.bits[313]._super._super * Val(2));
  Val x1584 = (arg2.bits[314]._super._super + x940);
  Val x1585 = (arg2.bits[314]._super._super * Val(2));
  Val x1586 = (arg2.bits[315]._super._super + x943);
  Val x1587 = (arg2.bits[315]._super._super * Val(2));
  Val x1588 = (arg2.bits[316]._super._super + x946);
  Val x1589 = (arg2.bits[316]._super._super * Val(2));
  Val x1590 = (arg2.bits[317]._super._super + x949);
  Val x1591 = (arg2.bits[317]._super._super * Val(2));
  Val x1592 = (arg2.bits[318]._super._super + x952);
  Val x1593 = (arg2.bits[318]._super._super * Val(2));
  Val x1594 = (arg2.bits[319]._super._super + x955);
  Val x1595 = (arg2.bits[319]._super._super * Val(2));
  Val x1596 = (arg1.bits[320]._super._super + x7);
  Val x1597 = (arg1.bits[320]._super._super * Val(2));
  Val x1598 = (arg1.bits[321]._super._super + x10);
  Val x1599 = (arg1.bits[321]._super._super * Val(2));
  Val x1600 = (arg1.bits[322]._super._super + x13);
  Val x1601 = (arg1.bits[322]._super._super * Val(2));
  Val x1602 = (arg1.bits[323]._super._super + x16);
  Val x1603 = (arg1.bits[323]._super._super * Val(2));
  Val x1604 = (arg1.bits[324]._super._super + x19);
  Val x1605 = (arg1.bits[324]._super._super * Val(2));
  Val x1606 = (arg1.bits[325]._super._super + x22);
  Val x1607 = (arg1.bits[325]._super._super * Val(2));
  Val x1608 = (arg1.bits[326]._super._super + x25);
  Val x1609 = (arg1.bits[326]._super._super * Val(2));
  Val x1610 = (arg1.bits[327]._super._super + x28);
  Val x1611 = (arg1.bits[327]._super._super * Val(2));
  Val x1612 = (arg1.bits[328]._super._super + x31);
  Val x1613 = (arg1.bits[328]._super._super * Val(2));
  Val x1614 = (arg1.bits[329]._super._super + x34);
  Val x1615 = (arg1.bits[329]._super._super * Val(2));
  Val x1616 = (arg1.bits[330]._super._super + x37);
  Val x1617 = (arg1.bits[330]._super._super * Val(2));
  Val x1618 = (arg1.bits[331]._super._super + x40);
  Val x1619 = (arg1.bits[331]._super._super * Val(2));
  Val x1620 = (arg1.bits[332]._super._super + x43);
  Val x1621 = (arg1.bits[332]._super._super * Val(2));
  Val x1622 = (arg1.bits[333]._super._super + x46);
  Val x1623 = (arg1.bits[333]._super._super * Val(2));
  Val x1624 = (arg1.bits[334]._super._super + x49);
  Val x1625 = (arg1.bits[334]._super._super * Val(2));
  Val x1626 = (arg1.bits[335]._super._super + x52);
  Val x1627 = (arg1.bits[335]._super._super * Val(2));
  Val x1628 = (arg1.bits[336]._super._super + x55);
  Val x1629 = (arg1.bits[336]._super._super * Val(2));
  Val x1630 = (arg1.bits[337]._super._super + x58);
  Val x1631 = (arg1.bits[337]._super._super * Val(2));
  Val x1632 = (arg1.bits[338]._super._super + x61);
  Val x1633 = (arg1.bits[338]._super._super * Val(2));
  Val x1634 = (arg1.bits[339]._super._super + x64);
  Val x1635 = (arg1.bits[339]._super._super * Val(2));
  Val x1636 = (arg1.bits[340]._super._super + x67);
  Val x1637 = (arg1.bits[340]._super._super * Val(2));
  Val x1638 = (arg1.bits[341]._super._super + x70);
  Val x1639 = (arg1.bits[341]._super._super * Val(2));
  Val x1640 = (arg1.bits[342]._super._super + x73);
  Val x1641 = (arg1.bits[342]._super._super * Val(2));
  Val x1642 = (arg1.bits[343]._super._super + x76);
  Val x1643 = (arg1.bits[343]._super._super * Val(2));
  Val x1644 = (arg1.bits[344]._super._super + x79);
  Val x1645 = (arg1.bits[344]._super._super * Val(2));
  Val x1646 = (arg1.bits[345]._super._super + x82);
  Val x1647 = (arg1.bits[345]._super._super * Val(2));
  Val x1648 = (arg1.bits[346]._super._super + x85);
  Val x1649 = (arg1.bits[346]._super._super * Val(2));
  Val x1650 = (arg1.bits[347]._super._super + x88);
  Val x1651 = (arg1.bits[347]._super._super * Val(2));
  Val x1652 = (arg1.bits[348]._super._super + x91);
  Val x1653 = (arg1.bits[348]._super._super * Val(2));
  Val x1654 = (arg2.bits[349]._super._super + x190);
  Val x1655 = (arg2.bits[349]._super._super * Val(2));
  Val x1656 = (arg2.bits[350]._super._super + x193);
  Val x1657 = (arg2.bits[350]._super._super * Val(2));
  Val x1658 = (arg2.bits[351]._super._super + x196);
  Val x1659 = (arg2.bits[351]._super._super * Val(2));
  Val x1660 = (arg1.bits[352]._super._super + x199);
  Val x1661 = (arg1.bits[352]._super._super * Val(2));
  Val x1662 = (arg1.bits[353]._super._super + x202);
  Val x1663 = (arg1.bits[353]._super._super * Val(2));
  Val x1664 = (arg1.bits[354]._super._super + x205);
  Val x1665 = (arg1.bits[354]._super._super * Val(2));
  Val x1666 = (arg1.bits[355]._super._super + x208);
  Val x1667 = (arg1.bits[355]._super._super * Val(2));
  Val x1668 = (arg1.bits[356]._super._super + x211);
  Val x1669 = (arg1.bits[356]._super._super * Val(2));
  Val x1670 = (arg1.bits[357]._super._super + x214);
  Val x1671 = (arg1.bits[357]._super._super * Val(2));
  Val x1672 = (arg1.bits[358]._super._super + x217);
  Val x1673 = (arg1.bits[358]._super._super * Val(2));
  Val x1674 = (arg1.bits[359]._super._super + x220);
  Val x1675 = (arg1.bits[359]._super._super * Val(2));
  Val x1676 = (arg1.bits[360]._super._super + x223);
  Val x1677 = (arg1.bits[360]._super._super * Val(2));
  Val x1678 = (arg1.bits[361]._super._super + x226);
  Val x1679 = (arg1.bits[361]._super._super * Val(2));
  Val x1680 = (arg1.bits[362]._super._super + x229);
  Val x1681 = (arg1.bits[362]._super._super * Val(2));
  Val x1682 = (arg1.bits[363]._super._super + x232);
  Val x1683 = (arg1.bits[363]._super._super * Val(2));
  Val x1684 = (arg1.bits[364]._super._super + x235);
  Val x1685 = (arg1.bits[364]._super._super * Val(2));
  Val x1686 = (arg1.bits[365]._super._super + x238);
  Val x1687 = (arg1.bits[365]._super._super * Val(2));
  Val x1688 = (arg1.bits[366]._super._super + x241);
  Val x1689 = (arg1.bits[366]._super._super * Val(2));
  Val x1690 = (arg1.bits[367]._super._super + x244);
  Val x1691 = (arg1.bits[367]._super._super * Val(2));
  Val x1692 = (arg1.bits[368]._super._super + x247);
  Val x1693 = (arg1.bits[368]._super._super * Val(2));
  Val x1694 = (arg1.bits[369]._super._super + x250);
  Val x1695 = (arg1.bits[369]._super._super * Val(2));
  Val x1696 = (arg1.bits[370]._super._super + x253);
  Val x1697 = (arg1.bits[370]._super._super * Val(2));
  Val x1698 = (arg1.bits[371]._super._super + x256);
  Val x1699 = (arg1.bits[371]._super._super * Val(2));
  Val x1700 = (arg1.bits[372]._super._super + x259);
  Val x1701 = (arg1.bits[372]._super._super * Val(2));
  Val x1702 = (arg1.bits[373]._super._super + x262);
  Val x1703 = (arg1.bits[373]._super._super * Val(2));
  Val x1704 = (arg2.bits[374]._super._super + x355);
  Val x1705 = (arg2.bits[374]._super._super * Val(2));
  Val x1706 = (arg2.bits[375]._super._super + x358);
  Val x1707 = (arg2.bits[375]._super._super * Val(2));
  Val x1708 = (arg2.bits[376]._super._super + x361);
  Val x1709 = (arg2.bits[376]._super._super * Val(2));
  Val x1710 = (arg2.bits[377]._super._super + x364);
  Val x1711 = (arg2.bits[377]._super._super * Val(2));
  Val x1712 = (arg2.bits[378]._super._super + x367);
  Val x1713 = (arg2.bits[378]._super._super * Val(2));
  Val x1714 = (arg2.bits[379]._super._super + x370);
  Val x1715 = (arg2.bits[379]._super._super * Val(2));
  Val x1716 = (arg2.bits[380]._super._super + x373);
  Val x1717 = (arg2.bits[380]._super._super * Val(2));
  Val x1718 = (arg2.bits[381]._super._super + x376);
  Val x1719 = (arg2.bits[381]._super._super * Val(2));
  Val x1720 = (arg2.bits[382]._super._super + x379);
  Val x1721 = (arg2.bits[382]._super._super * Val(2));
  Val x1722 = (arg2.bits[383]._super._super + x382);
  Val x1723 = (arg2.bits[383]._super._super * Val(2));
  Val x1724 = (arg1.bits[405]._super._super + x448);
  Val x1725 = (arg1.bits[405]._super._super * Val(2));
  Val x1726 = (arg1.bits[406]._super._super + x451);
  Val x1727 = (arg1.bits[406]._super._super * Val(2));
  Val x1728 = (arg1.bits[407]._super._super + x454);
  Val x1729 = (arg1.bits[407]._super._super * Val(2));
  Val x1730 = (arg1.bits[408]._super._super + x457);
  Val x1731 = (arg1.bits[408]._super._super * Val(2));
  Val x1732 = (arg1.bits[409]._super._super + x460);
  Val x1733 = (arg1.bits[409]._super._super * Val(2));
  Val x1734 = (arg1.bits[410]._super._super + x463);
  Val x1735 = (arg1.bits[410]._super._super * Val(2));
  Val x1736 = (arg1.bits[411]._super._super + x466);
  Val x1737 = (arg1.bits[411]._super._super * Val(2));
  Val x1738 = (arg1.bits[412]._super._super + x469);
  Val x1739 = (arg1.bits[412]._super._super * Val(2));
  Val x1740 = (arg1.bits[413]._super._super + x472);
  Val x1741 = (arg1.bits[413]._super._super * Val(2));
  Val x1742 = (arg1.bits[414]._super._super + x475);
  Val x1743 = (arg1.bits[414]._super._super * Val(2));
  Val x1744 = (arg1.bits[415]._super._super + x478);
  Val x1745 = (arg1.bits[415]._super._super * Val(2));
  Val x1746 = (arg2.bits[384]._super._super + x481);
  Val x1747 = (arg2.bits[384]._super._super * Val(2));
  Val x1748 = (arg2.bits[385]._super._super + x484);
  Val x1749 = (arg2.bits[385]._super._super * Val(2));
  Val x1750 = (arg2.bits[386]._super._super + x487);
  Val x1751 = (arg2.bits[386]._super._super * Val(2));
  Val x1752 = (arg2.bits[387]._super._super + x490);
  Val x1753 = (arg2.bits[387]._super._super * Val(2));
  Val x1754 = (arg2.bits[388]._super._super + x493);
  Val x1755 = (arg2.bits[388]._super._super * Val(2));
  Val x1756 = (arg2.bits[389]._super._super + x496);
  Val x1757 = (arg2.bits[389]._super._super * Val(2));
  Val x1758 = (arg2.bits[390]._super._super + x499);
  Val x1759 = (arg2.bits[390]._super._super * Val(2));
  Val x1760 = (arg2.bits[391]._super._super + x502);
  Val x1761 = (arg2.bits[391]._super._super * Val(2));
  Val x1762 = (arg2.bits[392]._super._super + x505);
  Val x1763 = (arg2.bits[392]._super._super * Val(2));
  Val x1764 = (arg2.bits[393]._super._super + x508);
  Val x1765 = (arg2.bits[393]._super._super * Val(2));
  Val x1766 = (arg2.bits[394]._super._super + x511);
  Val x1767 = (arg2.bits[394]._super._super * Val(2));
  Val x1768 = (arg2.bits[395]._super._super + x514);
  Val x1769 = (arg2.bits[395]._super._super * Val(2));
  Val x1770 = (arg2.bits[396]._super._super + x517);
  Val x1771 = (arg2.bits[396]._super._super * Val(2));
  Val x1772 = (arg2.bits[397]._super._super + x520);
  Val x1773 = (arg2.bits[397]._super._super * Val(2));
  Val x1774 = (arg2.bits[398]._super._super + x523);
  Val x1775 = (arg2.bits[398]._super._super * Val(2));
  Val x1776 = (arg2.bits[399]._super._super + x526);
  Val x1777 = (arg2.bits[399]._super._super * Val(2));
  Val x1778 = (arg2.bits[400]._super._super + x529);
  Val x1779 = (arg2.bits[400]._super._super * Val(2));
  Val x1780 = (arg2.bits[401]._super._super + x532);
  Val x1781 = (arg2.bits[401]._super._super * Val(2));
  Val x1782 = (arg2.bits[402]._super._super + x535);
  Val x1783 = (arg2.bits[402]._super._super * Val(2));
  Val x1784 = (arg2.bits[403]._super._super + x538);
  Val x1785 = (arg2.bits[403]._super._super * Val(2));
  Val x1786 = (arg2.bits[404]._super._super + x541);
  Val x1787 = (arg2.bits[404]._super._super * Val(2));
  Val x1788 = (arg1.bits[416]._super._super + x577);
  Val x1789 = (arg1.bits[416]._super._super * Val(2));
  Val x1790 = (arg1.bits[417]._super._super + x580);
  Val x1791 = (arg1.bits[417]._super._super * Val(2));
  Val x1792 = (arg1.bits[418]._super._super + x583);
  Val x1793 = (arg1.bits[418]._super._super * Val(2));
  Val x1794 = (arg1.bits[419]._super._super + x586);
  Val x1795 = (arg1.bits[419]._super._super * Val(2));
  Val x1796 = (arg1.bits[420]._super._super + x589);
  Val x1797 = (arg1.bits[420]._super._super * Val(2));
  Val x1798 = (arg1.bits[421]._super._super + x592);
  Val x1799 = (arg1.bits[421]._super._super * Val(2));
  Val x1800 = (arg1.bits[422]._super._super + x595);
  Val x1801 = (arg1.bits[422]._super._super * Val(2));
  Val x1802 = (arg2.bits[423]._super._super + x694);
  Val x1803 = (arg2.bits[423]._super._super * Val(2));
  Val x1804 = (arg2.bits[424]._super._super + x697);
  Val x1805 = (arg2.bits[424]._super._super * Val(2));
  Val x1806 = (arg2.bits[425]._super._super + x700);
  Val x1807 = (arg2.bits[425]._super._super * Val(2));
  Val x1808 = (arg2.bits[426]._super._super + x703);
  Val x1809 = (arg2.bits[426]._super._super * Val(2));
  Val x1810 = (arg2.bits[427]._super._super + x706);
  Val x1811 = (arg2.bits[427]._super._super * Val(2));
  Val x1812 = (arg2.bits[428]._super._super + x709);
  Val x1813 = (arg2.bits[428]._super._super * Val(2));
  Val x1814 = (arg2.bits[429]._super._super + x712);
  Val x1815 = (arg2.bits[429]._super._super * Val(2));
  Val x1816 = (arg2.bits[430]._super._super + x715);
  Val x1817 = (arg2.bits[430]._super._super * Val(2));
  Val x1818 = (arg2.bits[431]._super._super + x718);
  Val x1819 = (arg2.bits[431]._super._super * Val(2));
  Val x1820 = (arg2.bits[432]._super._super + x721);
  Val x1821 = (arg2.bits[432]._super._super * Val(2));
  Val x1822 = (arg2.bits[433]._super._super + x724);
  Val x1823 = (arg2.bits[433]._super._super * Val(2));
  Val x1824 = (arg2.bits[434]._super._super + x727);
  Val x1825 = (arg2.bits[434]._super._super * Val(2));
  Val x1826 = (arg2.bits[435]._super._super + x730);
  Val x1827 = (arg2.bits[435]._super._super * Val(2));
  Val x1828 = (arg2.bits[436]._super._super + x733);
  Val x1829 = (arg2.bits[436]._super._super * Val(2));
  Val x1830 = (arg2.bits[437]._super._super + x736);
  Val x1831 = (arg2.bits[437]._super._super * Val(2));
  Val x1832 = (arg2.bits[438]._super._super + x739);
  Val x1833 = (arg2.bits[438]._super._super * Val(2));
  Val x1834 = (arg2.bits[439]._super._super + x742);
  Val x1835 = (arg2.bits[439]._super._super * Val(2));
  Val x1836 = (arg2.bits[440]._super._super + x745);
  Val x1837 = (arg2.bits[440]._super._super * Val(2));
  Val x1838 = (arg2.bits[441]._super._super + x748);
  Val x1839 = (arg2.bits[441]._super._super * Val(2));
  Val x1840 = (arg2.bits[442]._super._super + x751);
  Val x1841 = (arg2.bits[442]._super._super * Val(2));
  Val x1842 = (arg2.bits[443]._super._super + x754);
  Val x1843 = (arg2.bits[443]._super._super * Val(2));
  Val x1844 = (arg2.bits[444]._super._super + x757);
  Val x1845 = (arg2.bits[444]._super._super * Val(2));
  Val x1846 = (arg2.bits[445]._super._super + x760);
  Val x1847 = (arg2.bits[445]._super._super * Val(2));
  Val x1848 = (arg2.bits[446]._super._super + x763);
  Val x1849 = (arg2.bits[446]._super._super * Val(2));
  Val x1850 = (arg2.bits[447]._super._super + x766);
  Val x1851 = (arg2.bits[447]._super._super * Val(2));
  Val x1852 = (arg1.bits[473]._super._super + x841);
  Val x1853 = (arg1.bits[473]._super._super * Val(2));
  Val x1854 = (arg1.bits[474]._super._super + x844);
  Val x1855 = (arg1.bits[474]._super._super * Val(2));
  Val x1856 = (arg1.bits[475]._super._super + x847);
  Val x1857 = (arg1.bits[475]._super._super * Val(2));
  Val x1858 = (arg1.bits[476]._super._super + x850);
  Val x1859 = (arg1.bits[476]._super._super * Val(2));
  Val x1860 = (arg1.bits[477]._super._super + x853);
  Val x1861 = (arg1.bits[477]._super._super * Val(2));
  Val x1862 = (arg1.bits[478]._super._super + x856);
  Val x1863 = (arg1.bits[478]._super._super * Val(2));
  Val x1864 = (arg1.bits[479]._super._super + x859);
  Val x1865 = (arg1.bits[479]._super._super * Val(2));
  Val x1866 = (arg2.bits[448]._super._super + x862);
  Val x1867 = (arg2.bits[448]._super._super * Val(2));
  Val x1868 = (arg2.bits[449]._super._super + x865);
  Val x1869 = (arg2.bits[449]._super._super * Val(2));
  Val x1870 = (arg2.bits[450]._super._super + x868);
  Val x1871 = (arg2.bits[450]._super._super * Val(2));
  Val x1872 = (arg2.bits[451]._super._super + x871);
  Val x1873 = (arg2.bits[451]._super._super * Val(2));
  Val x1874 = (arg2.bits[452]._super._super + x874);
  Val x1875 = (arg2.bits[452]._super._super * Val(2));
  Val x1876 = (arg2.bits[453]._super._super + x877);
  Val x1877 = (arg2.bits[453]._super._super * Val(2));
  Val x1878 = (arg2.bits[454]._super._super + x880);
  Val x1879 = (arg2.bits[454]._super._super * Val(2));
  Val x1880 = (arg2.bits[455]._super._super + x883);
  Val x1881 = (arg2.bits[455]._super._super * Val(2));
  Val x1882 = (arg2.bits[456]._super._super + x886);
  Val x1883 = (arg2.bits[456]._super._super * Val(2));
  Val x1884 = (arg2.bits[457]._super._super + x889);
  Val x1885 = (arg2.bits[457]._super._super * Val(2));
  Val x1886 = (arg2.bits[458]._super._super + x892);
  Val x1887 = (arg2.bits[458]._super._super * Val(2));
  Val x1888 = (arg2.bits[459]._super._super + x895);
  Val x1889 = (arg2.bits[459]._super._super * Val(2));
  Val x1890 = (arg2.bits[460]._super._super + x898);
  Val x1891 = (arg2.bits[460]._super._super * Val(2));
  Val x1892 = (arg2.bits[461]._super._super + x901);
  Val x1893 = (arg2.bits[461]._super._super * Val(2));
  Val x1894 = (arg2.bits[462]._super._super + x904);
  Val x1895 = (arg2.bits[462]._super._super * Val(2));
  Val x1896 = (arg2.bits[463]._super._super + x907);
  Val x1897 = (arg2.bits[463]._super._super * Val(2));
  Val x1898 = (arg2.bits[464]._super._super + x910);
  Val x1899 = (arg2.bits[464]._super._super * Val(2));
  Val x1900 = (arg2.bits[465]._super._super + x913);
  Val x1901 = (arg2.bits[465]._super._super * Val(2));
  Val x1902 = (arg2.bits[466]._super._super + x916);
  Val x1903 = (arg2.bits[466]._super._super * Val(2));
  Val x1904 = (arg2.bits[467]._super._super + x919);
  Val x1905 = (arg2.bits[467]._super._super * Val(2));
  Val x1906 = (arg2.bits[468]._super._super + x922);
  Val x1907 = (arg2.bits[468]._super._super * Val(2));
  Val x1908 = (arg2.bits[469]._super._super + x925);
  Val x1909 = (arg2.bits[469]._super._super * Val(2));
  Val x1910 = (arg2.bits[470]._super._super + x928);
  Val x1911 = (arg2.bits[470]._super._super * Val(2));
  Val x1912 = (arg2.bits[471]._super._super + x931);
  Val x1913 = (arg2.bits[471]._super._super * Val(2));
  Val x1914 = (arg2.bits[472]._super._super + x934);
  Val x1915 = (arg2.bits[472]._super._super * Val(2));
  Val x1916 = (arg1.bits[503]._super._super + x76);
  Val x1917 = (arg1.bits[503]._super._super * Val(2));
  Val x1918 = (arg1.bits[504]._super._super + x79);
  Val x1919 = (arg1.bits[504]._super._super * Val(2));
  Val x1920 = (arg1.bits[505]._super._super + x82);
  Val x1921 = (arg1.bits[505]._super._super * Val(2));
  Val x1922 = (arg1.bits[506]._super._super + x85);
  Val x1923 = (arg1.bits[506]._super._super * Val(2));
  Val x1924 = (arg1.bits[507]._super._super + x88);
  Val x1925 = (arg1.bits[507]._super._super * Val(2));
  Val x1926 = (arg1.bits[508]._super._super + x91);
  Val x1927 = (arg1.bits[508]._super._super * Val(2));
  Val x1928 = (arg1.bits[509]._super._super + x94);
  Val x1929 = (arg1.bits[509]._super._super * Val(2));
  Val x1930 = (arg1.bits[510]._super._super + x97);
  Val x1931 = (arg1.bits[510]._super._super * Val(2));
  Val x1932 = (arg1.bits[511]._super._super + x100);
  Val x1933 = (arg1.bits[511]._super._super * Val(2));
  Val x1934 = (arg2.bits[480]._super._super + x103);
  Val x1935 = (arg2.bits[480]._super._super * Val(2));
  Val x1936 = (arg2.bits[481]._super._super + x106);
  Val x1937 = (arg2.bits[481]._super._super * Val(2));
  Val x1938 = (arg2.bits[482]._super._super + x109);
  Val x1939 = (arg2.bits[482]._super._super * Val(2));
  Val x1940 = (arg2.bits[483]._super._super + x112);
  Val x1941 = (arg2.bits[483]._super._super * Val(2));
  Val x1942 = (arg2.bits[484]._super._super + x115);
  Val x1943 = (arg2.bits[484]._super._super * Val(2));
  Val x1944 = (arg2.bits[485]._super._super + x118);
  Val x1945 = (arg2.bits[485]._super._super * Val(2));
  Val x1946 = (arg2.bits[486]._super._super + x121);
  Val x1947 = (arg2.bits[486]._super._super * Val(2));
  Val x1948 = (arg2.bits[487]._super._super + x124);
  Val x1949 = (arg2.bits[487]._super._super * Val(2));
  Val x1950 = (arg2.bits[488]._super._super + x127);
  Val x1951 = (arg2.bits[488]._super._super * Val(2));
  Val x1952 = (arg2.bits[489]._super._super + x130);
  Val x1953 = (arg2.bits[489]._super._super * Val(2));
  Val x1954 = (arg2.bits[490]._super._super + x133);
  Val x1955 = (arg2.bits[490]._super._super * Val(2));
  Val x1956 = (arg2.bits[491]._super._super + x136);
  Val x1957 = (arg2.bits[491]._super._super * Val(2));
  Val x1958 = (arg2.bits[492]._super._super + x139);
  Val x1959 = (arg2.bits[492]._super._super * Val(2));
  Val x1960 = (arg2.bits[493]._super._super + x142);
  Val x1961 = (arg2.bits[493]._super._super * Val(2));
  Val x1962 = (arg2.bits[494]._super._super + x145);
  Val x1963 = (arg2.bits[494]._super._super * Val(2));
  Val x1964 = (arg2.bits[495]._super._super + x148);
  Val x1965 = (arg2.bits[495]._super._super * Val(2));
  Val x1966 = (arg2.bits[496]._super._super + x151);
  Val x1967 = (arg2.bits[496]._super._super * Val(2));
  Val x1968 = (arg2.bits[497]._super._super + x154);
  Val x1969 = (arg2.bits[497]._super._super * Val(2));
  Val x1970 = (arg2.bits[498]._super._super + x157);
  Val x1971 = (arg2.bits[498]._super._super * Val(2));
  Val x1972 = (arg2.bits[499]._super._super + x160);
  Val x1973 = (arg2.bits[499]._super._super * Val(2));
  Val x1974 = (arg2.bits[500]._super._super + x163);
  Val x1975 = (arg2.bits[500]._super._super * Val(2));
  Val x1976 = (arg2.bits[501]._super._super + x166);
  Val x1977 = (arg2.bits[501]._super._super * Val(2));
  Val x1978 = (arg2.bits[502]._super._super + x169);
  Val x1979 = (arg2.bits[502]._super._super * Val(2));
  Val x1980 = (arg1.bits[531]._super._super + x256);
  Val x1981 = (arg1.bits[531]._super._super * Val(2));
  Val x1982 = (arg1.bits[532]._super._super + x259);
  Val x1983 = (arg1.bits[532]._super._super * Val(2));
  Val x1984 = (arg1.bits[533]._super._super + x262);
  Val x1985 = (arg1.bits[533]._super._super * Val(2));
  Val x1986 = (arg1.bits[534]._super._super + x265);
  Val x1987 = (arg1.bits[534]._super._super * Val(2));
  Val x1988 = (arg1.bits[535]._super._super + x268);
  Val x1989 = (arg1.bits[535]._super._super * Val(2));
  Val x1990 = (arg1.bits[536]._super._super + x271);
  Val x1991 = (arg1.bits[536]._super._super * Val(2));
  Val x1992 = (arg1.bits[537]._super._super + x274);
  Val x1993 = (arg1.bits[537]._super._super * Val(2));
  Val x1994 = (arg1.bits[538]._super._super + x277);
  Val x1995 = (arg1.bits[538]._super._super * Val(2));
  Val x1996 = (arg1.bits[539]._super._super + x280);
  Val x1997 = (arg1.bits[539]._super._super * Val(2));
  Val x1998 = (arg1.bits[540]._super._super + x283);
  Val x1999 = (arg1.bits[540]._super._super * Val(2));
  Val x2000 = (arg1.bits[541]._super._super + x286);
  Val x2001 = (arg1.bits[541]._super._super * Val(2));
  Val x2002 = (arg1.bits[542]._super._super + x289);
  Val x2003 = (arg1.bits[542]._super._super * Val(2));
  Val x2004 = (arg1.bits[543]._super._super + x292);
  Val x2005 = (arg1.bits[543]._super._super * Val(2));
  Val x2006 = (arg2.bits[512]._super._super + x295);
  Val x2007 = (arg2.bits[512]._super._super * Val(2));
  Val x2008 = (arg2.bits[513]._super._super + x298);
  Val x2009 = (arg2.bits[513]._super._super * Val(2));
  Val x2010 = (arg2.bits[514]._super._super + x301);
  Val x2011 = (arg2.bits[514]._super._super * Val(2));
  Val x2012 = (arg2.bits[515]._super._super + x304);
  Val x2013 = (arg2.bits[515]._super._super * Val(2));
  Val x2014 = (arg2.bits[516]._super._super + x307);
  Val x2015 = (arg2.bits[516]._super._super * Val(2));
  Val x2016 = (arg2.bits[517]._super._super + x310);
  Val x2017 = (arg2.bits[517]._super._super * Val(2));
  Val x2018 = (arg2.bits[518]._super._super + x313);
  Val x2019 = (arg2.bits[518]._super._super * Val(2));
  Val x2020 = (arg2.bits[519]._super._super + x316);
  Val x2021 = (arg2.bits[519]._super._super * Val(2));
  Val x2022 = (arg2.bits[520]._super._super + x319);
  Val x2023 = (arg2.bits[520]._super._super * Val(2));
  Val x2024 = (arg2.bits[521]._super._super + x322);
  Val x2025 = (arg2.bits[521]._super._super * Val(2));
  Val x2026 = (arg2.bits[522]._super._super + x325);
  Val x2027 = (arg2.bits[522]._super._super * Val(2));
  Val x2028 = (arg2.bits[523]._super._super + x328);
  Val x2029 = (arg2.bits[523]._super._super * Val(2));
  Val x2030 = (arg2.bits[524]._super._super + x331);
  Val x2031 = (arg2.bits[524]._super._super * Val(2));
  Val x2032 = (arg2.bits[525]._super._super + x334);
  Val x2033 = (arg2.bits[525]._super._super * Val(2));
  Val x2034 = (arg2.bits[526]._super._super + x337);
  Val x2035 = (arg2.bits[526]._super._super * Val(2));
  Val x2036 = (arg2.bits[527]._super._super + x340);
  Val x2037 = (arg2.bits[527]._super._super * Val(2));
  Val x2038 = (arg2.bits[528]._super._super + x343);
  Val x2039 = (arg2.bits[528]._super._super * Val(2));
  Val x2040 = (arg2.bits[529]._super._super + x346);
  Val x2041 = (arg2.bits[529]._super._super * Val(2));
  Val x2042 = (arg2.bits[530]._super._super + x349);
  Val x2043 = (arg2.bits[530]._super._super * Val(2));
  Val x2044 = (arg1.bits[544]._super._super + x385);
  Val x2045 = (arg1.bits[544]._super._super * Val(2));
  Val x2046 = (arg1.bits[545]._super._super + x388);
  Val x2047 = (arg1.bits[545]._super._super * Val(2));
  Val x2048 = (arg1.bits[546]._super._super + x391);
  Val x2049 = (arg1.bits[546]._super._super * Val(2));
  Val x2050 = (arg1.bits[547]._super._super + x394);
  Val x2051 = (arg1.bits[547]._super._super * Val(2));
  Val x2052 = (arg1.bits[548]._super._super + x397);
  Val x2053 = (arg1.bits[548]._super._super * Val(2));
  Val x2054 = (arg1.bits[549]._super._super + x400);
  Val x2055 = (arg1.bits[549]._super._super * Val(2));
  Val x2056 = (arg1.bits[550]._super._super + x403);
  Val x2057 = (arg1.bits[550]._super._super * Val(2));
  Val x2058 = (arg1.bits[551]._super._super + x406);
  Val x2059 = (arg1.bits[551]._super._super * Val(2));
  Val x2060 = (arg1.bits[552]._super._super + x409);
  Val x2061 = (arg1.bits[552]._super._super * Val(2));
  Val x2062 = (arg1.bits[553]._super._super + x412);
  Val x2063 = (arg1.bits[553]._super._super * Val(2));
  Val x2064 = (arg1.bits[554]._super._super + x415);
  Val x2065 = (arg1.bits[554]._super._super * Val(2));
  Val x2066 = (arg1.bits[555]._super._super + x418);
  Val x2067 = (arg1.bits[555]._super._super * Val(2));
  Val x2068 = (arg1.bits[556]._super._super + x421);
  Val x2069 = (arg1.bits[556]._super._super * Val(2));
  Val x2070 = (arg1.bits[557]._super._super + x424);
  Val x2071 = (arg1.bits[557]._super._super * Val(2));
  Val x2072 = (arg1.bits[558]._super._super + x427);
  Val x2073 = (arg1.bits[558]._super._super * Val(2));
  Val x2074 = (arg1.bits[559]._super._super + x430);
  Val x2075 = (arg1.bits[559]._super._super * Val(2));
  Val x2076 = (arg1.bits[560]._super._super + x433);
  Val x2077 = (arg1.bits[560]._super._super * Val(2));
  Val x2078 = (arg2.bits[561]._super._super + x532);
  Val x2079 = (arg2.bits[561]._super._super * Val(2));
  Val x2080 = (arg2.bits[562]._super._super + x535);
  Val x2081 = (arg2.bits[562]._super._super * Val(2));
  Val x2082 = (arg2.bits[563]._super._super + x538);
  Val x2083 = (arg2.bits[563]._super._super * Val(2));
  Val x2084 = (arg2.bits[564]._super._super + x541);
  Val x2085 = (arg2.bits[564]._super._super * Val(2));
  Val x2086 = (arg2.bits[565]._super._super + x544);
  Val x2087 = (arg2.bits[565]._super._super * Val(2));
  Val x2088 = (arg2.bits[566]._super._super + x547);
  Val x2089 = (arg2.bits[566]._super._super * Val(2));
  Val x2090 = (arg2.bits[567]._super._super + x550);
  Val x2091 = (arg2.bits[567]._super._super * Val(2));
  Val x2092 = (arg2.bits[568]._super._super + x553);
  Val x2093 = (arg2.bits[568]._super._super * Val(2));
  Val x2094 = (arg2.bits[569]._super._super + x556);
  Val x2095 = (arg2.bits[569]._super._super * Val(2));
  Val x2096 = (arg2.bits[570]._super._super + x559);
  Val x2097 = (arg2.bits[570]._super._super * Val(2));
  Val x2098 = (arg2.bits[571]._super._super + x562);
  Val x2099 = (arg2.bits[571]._super._super * Val(2));
  Val x2100 = (arg2.bits[572]._super._super + x565);
  Val x2101 = (arg2.bits[572]._super._super * Val(2));
  Val x2102 = (arg2.bits[573]._super._super + x568);
  Val x2103 = (arg2.bits[573]._super._super * Val(2));
  Val x2104 = (arg2.bits[574]._super._super + x571);
  Val x2105 = (arg2.bits[574]._super._super * Val(2));
  Val x2106 = (arg2.bits[575]._super._super + x574);
  Val x2107 = (arg2.bits[575]._super._super * Val(2));
  Val x2108 = (arg1.bits[576]._super._super + x577);
  Val x2109 = (arg1.bits[576]._super._super * Val(2));
  Val x2110 = (arg1.bits[577]._super._super + x580);
  Val x2111 = (arg1.bits[577]._super._super * Val(2));
  Val x2112 = (arg1.bits[578]._super._super + x583);
  Val x2113 = (arg1.bits[578]._super._super * Val(2));
  Val x2114 = (arg1.bits[579]._super._super + x586);
  Val x2115 = (arg1.bits[579]._super._super * Val(2));
  Val x2116 = (arg1.bits[580]._super._super + x589);
  Val x2117 = (arg1.bits[580]._super._super * Val(2));
  Val x2118 = (arg1.bits[581]._super._super + x592);
  Val x2119 = (arg1.bits[581]._super._super * Val(2));
  Val x2120 = (arg1.bits[582]._super._super + x595);
  Val x2121 = (arg1.bits[582]._super._super * Val(2));
  Val x2122 = (arg1.bits[583]._super._super + x598);
  Val x2123 = (arg1.bits[583]._super._super * Val(2));
  Val x2124 = (arg1.bits[584]._super._super + x601);
  Val x2125 = (arg1.bits[584]._super._super * Val(2));
  Val x2126 = (arg1.bits[585]._super._super + x604);
  Val x2127 = (arg1.bits[585]._super._super * Val(2));
  Val x2128 = (arg1.bits[586]._super._super + x607);
  Val x2129 = (arg1.bits[586]._super._super * Val(2));
  Val x2130 = (arg2.bits[587]._super._super + x706);
  Val x2131 = (arg2.bits[587]._super._super * Val(2));
  Val x2132 = (arg2.bits[588]._super._super + x709);
  Val x2133 = (arg2.bits[588]._super._super * Val(2));
  Val x2134 = (arg2.bits[589]._super._super + x712);
  Val x2135 = (arg2.bits[589]._super._super * Val(2));
  Val x2136 = (arg2.bits[590]._super._super + x715);
  Val x2137 = (arg2.bits[590]._super._super * Val(2));
  Val x2138 = (arg2.bits[591]._super._super + x718);
  Val x2139 = (arg2.bits[591]._super._super * Val(2));
  Val x2140 = (arg2.bits[592]._super._super + x721);
  Val x2141 = (arg2.bits[592]._super._super * Val(2));
  Val x2142 = (arg2.bits[593]._super._super + x724);
  Val x2143 = (arg2.bits[593]._super._super * Val(2));
  Val x2144 = (arg2.bits[594]._super._super + x727);
  Val x2145 = (arg2.bits[594]._super._super * Val(2));
  Val x2146 = (arg2.bits[595]._super._super + x730);
  Val x2147 = (arg2.bits[595]._super._super * Val(2));
  Val x2148 = (arg2.bits[596]._super._super + x733);
  Val x2149 = (arg2.bits[596]._super._super * Val(2));
  Val x2150 = (arg2.bits[597]._super._super + x736);
  Val x2151 = (arg2.bits[597]._super._super * Val(2));
  Val x2152 = (arg2.bits[598]._super._super + x739);
  Val x2153 = (arg2.bits[598]._super._super * Val(2));
  Val x2154 = (arg2.bits[599]._super._super + x742);
  Val x2155 = (arg2.bits[599]._super._super * Val(2));
  Val x2156 = (arg2.bits[600]._super._super + x745);
  Val x2157 = (arg2.bits[600]._super._super * Val(2));
  Val x2158 = (arg2.bits[601]._super._super + x748);
  Val x2159 = (arg2.bits[601]._super._super * Val(2));
  Val x2160 = (arg2.bits[602]._super._super + x751);
  Val x2161 = (arg2.bits[602]._super._super * Val(2));
  Val x2162 = (arg2.bits[603]._super._super + x754);
  Val x2163 = (arg2.bits[603]._super._super * Val(2));
  Val x2164 = (arg2.bits[604]._super._super + x757);
  Val x2165 = (arg2.bits[604]._super._super * Val(2));
  Val x2166 = (arg2.bits[605]._super._super + x760);
  Val x2167 = (arg2.bits[605]._super._super * Val(2));
  Val x2168 = (arg2.bits[606]._super._super + x763);
  Val x2169 = (arg2.bits[606]._super._super * Val(2));
  Val x2170 = (arg2.bits[607]._super._super + x766);
  Val x2171 = (arg2.bits[607]._super._super * Val(2));
  Val x2172 = (arg1.bits[608]._super._super + x769);
  Val x2173 = (arg1.bits[608]._super._super * Val(2));
  Val x2174 = (arg1.bits[609]._super._super + x772);
  Val x2175 = (arg1.bits[609]._super._super * Val(2));
  Val x2176 = (arg1.bits[610]._super._super + x775);
  Val x2177 = (arg1.bits[610]._super._super * Val(2));
  Val x2178 = (arg1.bits[611]._super._super + x778);
  Val x2179 = (arg1.bits[611]._super._super * Val(2));
  Val x2180 = (arg1.bits[612]._super._super + x781);
  Val x2181 = (arg1.bits[612]._super._super * Val(2));
  Val x2182 = (arg1.bits[613]._super._super + x784);
  Val x2183 = (arg1.bits[613]._super._super * Val(2));
  Val x2184 = (arg1.bits[614]._super._super + x787);
  Val x2185 = (arg1.bits[614]._super._super * Val(2));
  Val x2186 = (arg1.bits[615]._super._super + x790);
  Val x2187 = (arg1.bits[615]._super._super * Val(2));
  Val x2188 = (arg1.bits[616]._super._super + x793);
  Val x2189 = (arg1.bits[616]._super._super * Val(2));
  Val x2190 = (arg1.bits[617]._super._super + x796);
  Val x2191 = (arg1.bits[617]._super._super * Val(2));
  Val x2192 = (arg1.bits[618]._super._super + x799);
  Val x2193 = (arg1.bits[618]._super._super * Val(2));
  Val x2194 = (arg1.bits[619]._super._super + x802);
  Val x2195 = (arg1.bits[619]._super._super * Val(2));
  Val x2196 = (arg1.bits[620]._super._super + x805);
  Val x2197 = (arg1.bits[620]._super._super * Val(2));
  Val x2198 = (arg1.bits[621]._super._super + x808);
  Val x2199 = (arg1.bits[621]._super._super * Val(2));
  Val x2200 = (arg1.bits[622]._super._super + x811);
  Val x2201 = (arg1.bits[622]._super._super * Val(2));
  Val x2202 = (arg1.bits[623]._super._super + x814);
  Val x2203 = (arg1.bits[623]._super._super * Val(2));
  Val x2204 = (arg1.bits[624]._super._super + x817);
  Val x2205 = (arg1.bits[624]._super._super * Val(2));
  Val x2206 = (arg1.bits[625]._super._super + x820);
  Val x2207 = (arg1.bits[625]._super._super * Val(2));
  Val x2208 = (arg1.bits[626]._super._super + x823);
  Val x2209 = (arg1.bits[626]._super._super * Val(2));
  Val x2210 = (arg1.bits[627]._super._super + x826);
  Val x2211 = (arg1.bits[627]._super._super * Val(2));
  Val x2212 = (arg1.bits[628]._super._super + x829);
  Val x2213 = (arg1.bits[628]._super._super * Val(2));
  Val x2214 = (arg1.bits[629]._super._super + x832);
  Val x2215 = (arg1.bits[629]._super._super * Val(2));
  Val x2216 = (arg1.bits[630]._super._super + x835);
  Val x2217 = (arg1.bits[630]._super._super * Val(2));
  Val x2218 = (arg1.bits[631]._super._super + x838);
  Val x2219 = (arg1.bits[631]._super._super * Val(2));
  Val x2220 = (arg2.bits[632]._super._super + x934);
  Val x2221 = (arg2.bits[632]._super._super * Val(2));
  Val x2222 = (arg2.bits[633]._super._super + x937);
  Val x2223 = (arg2.bits[633]._super._super * Val(2));
  Val x2224 = (arg2.bits[634]._super._super + x940);
  Val x2225 = (arg2.bits[634]._super._super * Val(2));
  Val x2226 = (arg2.bits[635]._super._super + x943);
  Val x2227 = (arg2.bits[635]._super._super * Val(2));
  Val x2228 = (arg2.bits[636]._super._super + x946);
  Val x2229 = (arg2.bits[636]._super._super * Val(2));
  Val x2230 = (arg2.bits[637]._super._super + x949);
  Val x2231 = (arg2.bits[637]._super._super * Val(2));
  Val x2232 = (arg2.bits[638]._super._super + x952);
  Val x2233 = (arg2.bits[638]._super._super * Val(2));
  Val x2234 = (arg2.bits[639]._super._super + x955);
  Val x2235 = (arg2.bits[639]._super._super * Val(2));
  Val x2236 = (arg1.bits[640]._super._super + x7);
  Val x2237 = (arg1.bits[640]._super._super * Val(2));
  Val x2238 = (arg1.bits[641]._super._super + x10);
  Val x2239 = (arg1.bits[641]._super._super * Val(2));
  Val x2240 = (arg1.bits[642]._super._super + x13);
  Val x2241 = (arg1.bits[642]._super._super * Val(2));
  Val x2242 = (arg1.bits[643]._super._super + x16);
  Val x2243 = (arg1.bits[643]._super._super * Val(2));
  Val x2244 = (arg1.bits[644]._super._super + x19);
  Val x2245 = (arg1.bits[644]._super._super * Val(2));
  Val x2246 = (arg1.bits[645]._super._super + x22);
  Val x2247 = (arg1.bits[645]._super._super * Val(2));
  Val x2248 = (arg1.bits[646]._super._super + x25);
  Val x2249 = (arg1.bits[646]._super._super * Val(2));
  Val x2250 = (arg1.bits[647]._super._super + x28);
  Val x2251 = (arg1.bits[647]._super._super * Val(2));
  Val x2252 = (arg1.bits[648]._super._super + x31);
  Val x2253 = (arg1.bits[648]._super._super * Val(2));
  Val x2254 = (arg1.bits[649]._super._super + x34);
  Val x2255 = (arg1.bits[649]._super._super * Val(2));
  Val x2256 = (arg1.bits[650]._super._super + x37);
  Val x2257 = (arg1.bits[650]._super._super * Val(2));
  Val x2258 = (arg1.bits[651]._super._super + x40);
  Val x2259 = (arg1.bits[651]._super._super * Val(2));
  Val x2260 = (arg1.bits[652]._super._super + x43);
  Val x2261 = (arg1.bits[652]._super._super * Val(2));
  Val x2262 = (arg1.bits[653]._super._super + x46);
  Val x2263 = (arg1.bits[653]._super._super * Val(2));
  Val x2264 = (arg2.bits[654]._super._super + x145);
  Val x2265 = (arg2.bits[654]._super._super * Val(2));
  Val x2266 = (arg2.bits[655]._super._super + x148);
  Val x2267 = (arg2.bits[655]._super._super * Val(2));
  Val x2268 = (arg2.bits[656]._super._super + x151);
  Val x2269 = (arg2.bits[656]._super._super * Val(2));
  Val x2270 = (arg2.bits[657]._super._super + x154);
  Val x2271 = (arg2.bits[657]._super._super * Val(2));
  Val x2272 = (arg2.bits[658]._super._super + x157);
  Val x2273 = (arg2.bits[658]._super._super * Val(2));
  Val x2274 = (arg2.bits[659]._super._super + x160);
  Val x2275 = (arg2.bits[659]._super._super * Val(2));
  Val x2276 = (arg2.bits[660]._super._super + x163);
  Val x2277 = (arg2.bits[660]._super._super * Val(2));
  Val x2278 = (arg2.bits[661]._super._super + x166);
  Val x2279 = (arg2.bits[661]._super._super * Val(2));
  Val x2280 = (arg2.bits[662]._super._super + x169);
  Val x2281 = (arg2.bits[662]._super._super * Val(2));
  Val x2282 = (arg2.bits[663]._super._super + x172);
  Val x2283 = (arg2.bits[663]._super._super * Val(2));
  Val x2284 = (arg2.bits[664]._super._super + x175);
  Val x2285 = (arg2.bits[664]._super._super * Val(2));
  Val x2286 = (arg2.bits[665]._super._super + x178);
  Val x2287 = (arg2.bits[665]._super._super * Val(2));
  Val x2288 = (arg2.bits[666]._super._super + x181);
  Val x2289 = (arg2.bits[666]._super._super * Val(2));
  Val x2290 = (arg2.bits[667]._super._super + x184);
  Val x2291 = (arg2.bits[667]._super._super * Val(2));
  Val x2292 = (arg2.bits[668]._super._super + x187);
  Val x2293 = (arg2.bits[668]._super._super * Val(2));
  Val x2294 = (arg2.bits[669]._super._super + x190);
  Val x2295 = (arg2.bits[669]._super._super * Val(2));
  Val x2296 = (arg2.bits[670]._super._super + x193);
  Val x2297 = (arg2.bits[670]._super._super * Val(2));
  Val x2298 = (arg2.bits[671]._super._super + x196);
  Val x2299 = (arg2.bits[671]._super._super * Val(2));
  Val x2300 = (arg1.bits[672]._super._super + x199);
  Val x2301 = (arg1.bits[672]._super._super * Val(2));
  Val x2302 = (arg1.bits[673]._super._super + x202);
  Val x2303 = (arg1.bits[673]._super._super * Val(2));
  Val x2304 = (arg1.bits[674]._super._super + x205);
  Val x2305 = (arg1.bits[674]._super._super * Val(2));
  Val x2306 = (arg1.bits[675]._super._super + x208);
  Val x2307 = (arg1.bits[675]._super._super * Val(2));
  Val x2308 = (arg1.bits[676]._super._super + x211);
  Val x2309 = (arg1.bits[676]._super._super * Val(2));
  Val x2310 = (arg1.bits[677]._super._super + x214);
  Val x2311 = (arg1.bits[677]._super._super * Val(2));
  Val x2312 = (arg1.bits[678]._super._super + x217);
  Val x2313 = (arg1.bits[678]._super._super * Val(2));
  Val x2314 = (arg1.bits[679]._super._super + x220);
  Val x2315 = (arg1.bits[679]._super._super * Val(2));
  Val x2316 = (arg1.bits[680]._super._super + x223);
  Val x2317 = (arg1.bits[680]._super._super * Val(2));
  Val x2318 = (arg1.bits[681]._super._super + x226);
  Val x2319 = (arg1.bits[681]._super._super * Val(2));
  Val x2320 = (arg1.bits[682]._super._super + x229);
  Val x2321 = (arg1.bits[682]._super._super * Val(2));
  Val x2322 = (arg1.bits[683]._super._super + x232);
  Val x2323 = (arg1.bits[683]._super._super * Val(2));
  Val x2324 = (arg1.bits[684]._super._super + x235);
  Val x2325 = (arg1.bits[684]._super._super * Val(2));
  Val x2326 = (arg1.bits[685]._super._super + x238);
  Val x2327 = (arg1.bits[685]._super._super * Val(2));
  Val x2328 = (arg1.bits[686]._super._super + x241);
  Val x2329 = (arg1.bits[686]._super._super * Val(2));
  Val x2330 = (arg1.bits[687]._super._super + x244);
  Val x2331 = (arg1.bits[687]._super._super * Val(2));
  Val x2332 = (arg1.bits[688]._super._super + x247);
  Val x2333 = (arg1.bits[688]._super._super * Val(2));
  Val x2334 = (arg1.bits[689]._super._super + x250);
  Val x2335 = (arg1.bits[689]._super._super * Val(2));
  Val x2336 = (arg1.bits[690]._super._super + x253);
  Val x2337 = (arg1.bits[690]._super._super * Val(2));
  Val x2338 = (arg1.bits[691]._super._super + x256);
  Val x2339 = (arg1.bits[691]._super._super * Val(2));
  Val x2340 = (arg1.bits[692]._super._super + x259);
  Val x2341 = (arg1.bits[692]._super._super * Val(2));
  Val x2342 = (arg1.bits[693]._super._super + x262);
  Val x2343 = (arg1.bits[693]._super._super * Val(2));
  Val x2344 = (arg1.bits[694]._super._super + x265);
  Val x2345 = (arg1.bits[694]._super._super * Val(2));
  Val x2346 = (arg1.bits[695]._super._super + x268);
  Val x2347 = (arg1.bits[695]._super._super * Val(2));
  Val x2348 = (arg1.bits[696]._super._super + x271);
  Val x2349 = (arg1.bits[696]._super._super * Val(2));
  Val x2350 = (arg1.bits[697]._super._super + x274);
  Val x2351 = (arg1.bits[697]._super._super * Val(2));
  Val x2352 = (arg1.bits[698]._super._super + x277);
  Val x2353 = (arg1.bits[698]._super._super * Val(2));
  Val x2354 = (arg1.bits[699]._super._super + x280);
  Val x2355 = (arg1.bits[699]._super._super * Val(2));
  Val x2356 = (arg1.bits[700]._super._super + x283);
  Val x2357 = (arg1.bits[700]._super._super * Val(2));
  Val x2358 = (arg1.bits[701]._super._super + x286);
  Val x2359 = (arg1.bits[701]._super._super * Val(2));
  Val x2360 = (arg2.bits[702]._super._super + x379);
  Val x2361 = (arg2.bits[702]._super._super * Val(2));
  Val x2362 = (arg2.bits[703]._super._super + x382);
  Val x2363 = (arg2.bits[703]._super._super * Val(2));
  Val x2364 = (arg1.bits[707]._super._super + x394);
  Val x2365 = (arg1.bits[707]._super._super * Val(2));
  Val x2366 = (arg1.bits[708]._super._super + x397);
  Val x2367 = (arg1.bits[708]._super._super * Val(2));
  Val x2368 = (arg1.bits[709]._super._super + x400);
  Val x2369 = (arg1.bits[709]._super._super * Val(2));
  Val x2370 = (arg1.bits[710]._super._super + x403);
  Val x2371 = (arg1.bits[710]._super._super * Val(2));
  Val x2372 = (arg1.bits[711]._super._super + x406);
  Val x2373 = (arg1.bits[711]._super._super * Val(2));
  Val x2374 = (arg1.bits[712]._super._super + x409);
  Val x2375 = (arg1.bits[712]._super._super * Val(2));
  Val x2376 = (arg1.bits[713]._super._super + x412);
  Val x2377 = (arg1.bits[713]._super._super * Val(2));
  Val x2378 = (arg1.bits[714]._super._super + x415);
  Val x2379 = (arg1.bits[714]._super._super * Val(2));
  Val x2380 = (arg1.bits[715]._super._super + x418);
  Val x2381 = (arg1.bits[715]._super._super * Val(2));
  Val x2382 = (arg1.bits[716]._super._super + x421);
  Val x2383 = (arg1.bits[716]._super._super * Val(2));
  Val x2384 = (arg1.bits[717]._super._super + x424);
  Val x2385 = (arg1.bits[717]._super._super * Val(2));
  Val x2386 = (arg1.bits[718]._super._super + x427);
  Val x2387 = (arg1.bits[718]._super._super * Val(2));
  Val x2388 = (arg1.bits[719]._super._super + x430);
  Val x2389 = (arg1.bits[719]._super._super * Val(2));
  Val x2390 = (arg1.bits[720]._super._super + x433);
  Val x2391 = (arg1.bits[720]._super._super * Val(2));
  Val x2392 = (arg1.bits[721]._super._super + x436);
  Val x2393 = (arg1.bits[721]._super._super * Val(2));
  Val x2394 = (arg1.bits[722]._super._super + x439);
  Val x2395 = (arg1.bits[722]._super._super * Val(2));
  Val x2396 = (arg1.bits[723]._super._super + x442);
  Val x2397 = (arg1.bits[723]._super._super * Val(2));
  Val x2398 = (arg1.bits[724]._super._super + x445);
  Val x2399 = (arg1.bits[724]._super._super * Val(2));
  Val x2400 = (arg1.bits[725]._super._super + x448);
  Val x2401 = (arg1.bits[725]._super._super * Val(2));
  Val x2402 = (arg1.bits[726]._super._super + x451);
  Val x2403 = (arg1.bits[726]._super._super * Val(2));
  Val x2404 = (arg1.bits[727]._super._super + x454);
  Val x2405 = (arg1.bits[727]._super._super * Val(2));
  Val x2406 = (arg1.bits[728]._super._super + x457);
  Val x2407 = (arg1.bits[728]._super._super * Val(2));
  Val x2408 = (arg1.bits[729]._super._super + x460);
  Val x2409 = (arg1.bits[729]._super._super * Val(2));
  Val x2410 = (arg1.bits[730]._super._super + x463);
  Val x2411 = (arg1.bits[730]._super._super * Val(2));
  Val x2412 = (arg1.bits[731]._super._super + x466);
  Val x2413 = (arg1.bits[731]._super._super * Val(2));
  Val x2414 = (arg1.bits[732]._super._super + x469);
  Val x2415 = (arg1.bits[732]._super._super * Val(2));
  Val x2416 = (arg1.bits[733]._super._super + x472);
  Val x2417 = (arg1.bits[733]._super._super * Val(2));
  Val x2418 = (arg1.bits[734]._super._super + x475);
  Val x2419 = (arg1.bits[734]._super._super * Val(2));
  Val x2420 = (arg1.bits[735]._super._super + x478);
  Val x2421 = (arg1.bits[735]._super._super * Val(2));
  Val x2422 = (arg2.bits[704]._super._super + x481);
  Val x2423 = (arg2.bits[704]._super._super * Val(2));
  Val x2424 = (arg2.bits[705]._super._super + x484);
  Val x2425 = (arg2.bits[705]._super._super * Val(2));
  Val x2426 = (arg2.bits[706]._super._super + x487);
  Val x2427 = (arg2.bits[706]._super._super * Val(2));
  Val x2428 = (arg1.bits[744]._super._super + x601);
  Val x2429 = (arg1.bits[744]._super._super * Val(2));
  Val x2430 = (arg1.bits[745]._super._super + x604);
  Val x2431 = (arg1.bits[745]._super._super * Val(2));
  Val x2432 = (arg1.bits[746]._super._super + x607);
  Val x2433 = (arg1.bits[746]._super._super * Val(2));
  Val x2434 = (arg1.bits[747]._super._super + x610);
  Val x2435 = (arg1.bits[747]._super._super * Val(2));
  Val x2436 = (arg1.bits[748]._super._super + x613);
  Val x2437 = (arg1.bits[748]._super._super * Val(2));
  Val x2438 = (arg1.bits[749]._super._super + x616);
  Val x2439 = (arg1.bits[749]._super._super * Val(2));
  Val x2440 = (arg1.bits[750]._super._super + x619);
  Val x2441 = (arg1.bits[750]._super._super * Val(2));
  Val x2442 = (arg1.bits[751]._super._super + x622);
  Val x2443 = (arg1.bits[751]._super._super * Val(2));
  Val x2444 = (arg1.bits[752]._super._super + x625);
  Val x2445 = (arg1.bits[752]._super._super * Val(2));
  Val x2446 = (arg1.bits[753]._super._super + x628);
  Val x2447 = (arg1.bits[753]._super._super * Val(2));
  Val x2448 = (arg1.bits[754]._super._super + x631);
  Val x2449 = (arg1.bits[754]._super._super * Val(2));
  Val x2450 = (arg1.bits[755]._super._super + x634);
  Val x2451 = (arg1.bits[755]._super._super * Val(2));
  Val x2452 = (arg1.bits[756]._super._super + x637);
  Val x2453 = (arg1.bits[756]._super._super * Val(2));
  Val x2454 = (arg1.bits[757]._super._super + x640);
  Val x2455 = (arg1.bits[757]._super._super * Val(2));
  Val x2456 = (arg1.bits[758]._super._super + x643);
  Val x2457 = (arg1.bits[758]._super._super * Val(2));
  Val x2458 = (arg1.bits[759]._super._super + x646);
  Val x2459 = (arg1.bits[759]._super._super * Val(2));
  Val x2460 = (arg1.bits[760]._super._super + x649);
  Val x2461 = (arg1.bits[760]._super._super * Val(2));
  Val x2462 = (arg1.bits[761]._super._super + x652);
  Val x2463 = (arg1.bits[761]._super._super * Val(2));
  Val x2464 = (arg1.bits[762]._super._super + x655);
  Val x2465 = (arg1.bits[762]._super._super * Val(2));
  Val x2466 = (arg1.bits[763]._super._super + x658);
  Val x2467 = (arg1.bits[763]._super._super * Val(2));
  Val x2468 = (arg1.bits[764]._super._super + x661);
  Val x2469 = (arg1.bits[764]._super._super * Val(2));
  Val x2470 = (arg1.bits[765]._super._super + x664);
  Val x2471 = (arg1.bits[765]._super._super * Val(2));
  Val x2472 = (arg1.bits[766]._super._super + x667);
  Val x2473 = (arg1.bits[766]._super._super * Val(2));
  Val x2474 = (arg1.bits[767]._super._super + x670);
  Val x2475 = (arg1.bits[767]._super._super * Val(2));
  Val x2476 = (arg2.bits[736]._super._super + x673);
  Val x2477 = (arg2.bits[736]._super._super * Val(2));
  Val x2478 = (arg2.bits[737]._super._super + x676);
  Val x2479 = (arg2.bits[737]._super._super * Val(2));
  Val x2480 = (arg2.bits[738]._super._super + x679);
  Val x2481 = (arg2.bits[738]._super._super * Val(2));
  Val x2482 = (arg2.bits[739]._super._super + x682);
  Val x2483 = (arg2.bits[739]._super._super * Val(2));
  Val x2484 = (arg2.bits[740]._super._super + x685);
  Val x2485 = (arg2.bits[740]._super._super * Val(2));
  Val x2486 = (arg2.bits[741]._super._super + x688);
  Val x2487 = (arg2.bits[741]._super._super * Val(2));
  Val x2488 = (arg2.bits[742]._super._super + x691);
  Val x2489 = (arg2.bits[742]._super._super * Val(2));
  Val x2490 = (arg2.bits[743]._super._super + x694);
  Val x2491 = (arg2.bits[743]._super._super * Val(2));
  Val x2492 = (arg1.bits[768]._super._super + x769);
  Val x2493 = (arg1.bits[768]._super._super * Val(2));
  Val x2494 = (arg1.bits[769]._super._super + x772);
  Val x2495 = (arg1.bits[769]._super._super * Val(2));
  Val x2496 = (arg1.bits[770]._super._super + x775);
  Val x2497 = (arg1.bits[770]._super._super * Val(2));
  Val x2498 = (arg1.bits[771]._super._super + x778);
  Val x2499 = (arg1.bits[771]._super._super * Val(2));
  Val x2500 = (arg1.bits[772]._super._super + x781);
  Val x2501 = (arg1.bits[772]._super._super * Val(2));
  Val x2502 = (arg1.bits[773]._super._super + x784);
  Val x2503 = (arg1.bits[773]._super._super * Val(2));
  Val x2504 = (arg1.bits[774]._super._super + x787);
  Val x2505 = (arg1.bits[774]._super._super * Val(2));
  Val x2506 = (arg1.bits[775]._super._super + x790);
  Val x2507 = (arg1.bits[775]._super._super * Val(2));
  Val x2508 = (arg1.bits[776]._super._super + x793);
  Val x2509 = (arg1.bits[776]._super._super * Val(2));
  Val x2510 = (arg1.bits[777]._super._super + x796);
  Val x2511 = (arg1.bits[777]._super._super * Val(2));
  Val x2512 = (arg1.bits[778]._super._super + x799);
  Val x2513 = (arg1.bits[778]._super._super * Val(2));
  Val x2514 = (arg1.bits[779]._super._super + x802);
  Val x2515 = (arg1.bits[779]._super._super * Val(2));
  Val x2516 = (arg1.bits[780]._super._super + x805);
  Val x2517 = (arg1.bits[780]._super._super * Val(2));
  Val x2518 = (arg1.bits[781]._super._super + x808);
  Val x2519 = (arg1.bits[781]._super._super * Val(2));
  Val x2520 = (arg1.bits[782]._super._super + x811);
  Val x2521 = (arg1.bits[782]._super._super * Val(2));
  Val x2522 = (arg1.bits[783]._super._super + x814);
  Val x2523 = (arg1.bits[783]._super._super * Val(2));
  Val x2524 = (arg1.bits[784]._super._super + x817);
  Val x2525 = (arg1.bits[784]._super._super * Val(2));
  Val x2526 = (arg1.bits[785]._super._super + x820);
  Val x2527 = (arg1.bits[785]._super._super * Val(2));
  Val x2528 = (arg2.bits[786]._super._super + x916);
  Val x2529 = (arg2.bits[786]._super._super * Val(2));
  Val x2530 = (arg2.bits[787]._super._super + x919);
  Val x2531 = (arg2.bits[787]._super._super * Val(2));
  Val x2532 = (arg2.bits[788]._super._super + x922);
  Val x2533 = (arg2.bits[788]._super._super * Val(2));
  Val x2534 = (arg2.bits[789]._super._super + x925);
  Val x2535 = (arg2.bits[789]._super._super * Val(2));
  Val x2536 = (arg2.bits[790]._super._super + x928);
  Val x2537 = (arg2.bits[790]._super._super * Val(2));
  Val x2538 = (arg2.bits[791]._super._super + x931);
  Val x2539 = (arg2.bits[791]._super._super * Val(2));
  Val x2540 = (arg2.bits[792]._super._super + x934);
  Val x2541 = (arg2.bits[792]._super._super * Val(2));
  Val x2542 = (arg2.bits[793]._super._super + x937);
  Val x2543 = (arg2.bits[793]._super._super * Val(2));
  Val x2544 = (arg2.bits[794]._super._super + x940);
  Val x2545 = (arg2.bits[794]._super._super * Val(2));
  Val x2546 = (arg2.bits[795]._super._super + x943);
  Val x2547 = (arg2.bits[795]._super._super * Val(2));
  Val x2548 = (arg2.bits[796]._super._super + x946);
  Val x2549 = (arg2.bits[796]._super._super * Val(2));
  Val x2550 = (arg2.bits[797]._super._super + x949);
  Val x2551 = (arg2.bits[797]._super._super * Val(2));
  Val x2552 = (arg2.bits[798]._super._super + x952);
  Val x2553 = (arg2.bits[798]._super._super * Val(2));
  Val x2554 = (arg2.bits[799]._super._super + x955);
  Val x2555 = (arg2.bits[799]._super._super * Val(2));
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:107)
  Val100Array x2556 = Val100Array{
      arg3.kflat[0]._super._super,  arg3.kflat[1]._super._super,  arg3.kflat[2]._super._super,
      arg3.kflat[3]._super._super,  arg3.kflat[4]._super._super,  arg3.kflat[5]._super._super,
      arg3.kflat[6]._super._super,  arg3.kflat[7]._super._super,  arg3.kflat[8]._super._super,
      arg3.kflat[9]._super._super,  arg3.kflat[10]._super._super, arg3.kflat[11]._super._super,
      arg3.kflat[12]._super._super, arg3.kflat[13]._super._super, arg3.kflat[14]._super._super,
      arg3.kflat[15]._super._super, arg3.kflat[16]._super._super, arg3.kflat[17]._super._super,
      arg3.kflat[18]._super._super, arg3.kflat[19]._super._super, arg3.kflat[20]._super._super,
      arg3.kflat[21]._super._super, arg3.kflat[22]._super._super, arg3.kflat[23]._super._super,
      arg3.kflat[24]._super._super, arg3.kflat[25]._super._super, arg3.kflat[26]._super._super,
      arg3.kflat[27]._super._super, arg3.kflat[28]._super._super, arg3.kflat[29]._super._super,
      arg3.kflat[30]._super._super, arg3.kflat[31]._super._super, arg3.kflat[32]._super._super,
      arg3.kflat[33]._super._super, arg3.kflat[34]._super._super, arg3.kflat[35]._super._super,
      arg3.kflat[36]._super._super, arg3.kflat[37]._super._super, arg3.kflat[38]._super._super,
      arg3.kflat[39]._super._super, arg3.kflat[40]._super._super, arg3.kflat[41]._super._super,
      arg3.kflat[42]._super._super, arg3.kflat[43]._super._super, arg3.kflat[44]._super._super,
      arg3.kflat[45]._super._super, arg3.kflat[46]._super._super, arg3.kflat[47]._super._super,
      arg3.kflat[48]._super._super, arg3.kflat[49]._super._super, arg3.kflat[50]._super._super,
      arg3.kflat[51]._super._super, arg3.kflat[52]._super._super, arg3.kflat[53]._super._super,
      arg3.kflat[54]._super._super, arg3.kflat[55]._super._super, arg3.kflat[56]._super._super,
      arg3.kflat[57]._super._super, arg3.kflat[58]._super._super, arg3.kflat[59]._super._super,
      arg3.kflat[60]._super._super, arg3.kflat[61]._super._super, arg3.kflat[62]._super._super,
      arg3.kflat[63]._super._super, arg3.kflat[64]._super._super, arg3.kflat[65]._super._super,
      arg3.kflat[66]._super._super, arg3.kflat[67]._super._super, arg3.kflat[68]._super._super,
      arg3.kflat[69]._super._super, arg3.kflat[70]._super._super, arg3.kflat[71]._super._super,
      arg3.kflat[72]._super._super, arg3.kflat[73]._super._super, arg3.kflat[74]._super._super,
      arg3.kflat[75]._super._super, arg3.kflat[76]._super._super, arg3.kflat[77]._super._super,
      arg3.kflat[78]._super._super, arg3.kflat[79]._super._super, arg3.kflat[80]._super._super,
      arg3.kflat[81]._super._super, arg3.kflat[82]._super._super, arg3.kflat[83]._super._super,
      arg3.kflat[84]._super._super, arg3.kflat[85]._super._super, arg3.kflat[86]._super._super,
      arg3.kflat[87]._super._super, arg3.kflat[88]._super._super, arg3.kflat[89]._super._super,
      arg3.kflat[90]._super._super, arg3.kflat[91]._super._super, arg3.kflat[92]._super._super,
      arg3.kflat[93]._super._super, arg3.kflat[94]._super._super, arg3.kflat[95]._super._super,
      arg3.kflat[96]._super._super, arg3.kflat[97]._super._super, arg3.kflat[98]._super._super,
      arg3.kflat[99]._super._super};
  Val16Array x2557 = Val16Array{arg3.sflat[0]._super,
                                arg3.sflat[1]._super,
                                arg3.sflat[2]._super,
                                arg3.sflat[3]._super,
                                arg3.sflat[4]._super,
                                arg3.sflat[5]._super,
                                arg3.sflat[6]._super,
                                arg3.sflat[7]._super,
                                arg3.sflat[8]._super,
                                arg3.sflat[9]._super,
                                arg3.sflat[10]._super,
                                arg3.sflat[11]._super,
                                arg3.sflat[12]._super,
                                arg3.sflat[13]._super,
                                arg3.sflat[14]._super,
                                arg3.sflat[15]._super};
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:111)
  TopStateStruct x2558 = exec_TopState(
      ctx,
      Val800Array{(x956 - (x957 * x7)),     (x958 - (x959 * x10)),    (x960 - (x961 * x13)),
                  (x962 - (x963 * x16)),    (x964 - (x965 * x19)),    (x966 - (x967 * x22)),
                  (x968 - (x969 * x25)),    (x970 - (x971 * x28)),    (x972 - (x973 * x31)),
                  (x974 - (x975 * x34)),    (x976 - (x977 * x37)),    (x978 - (x979 * x40)),
                  (x980 - (x981 * x43)),    (x982 - (x983 * x46)),    (x984 - (x985 * x49)),
                  (x986 - (x987 * x52)),    (x988 - (x989 * x55)),    (x990 - (x991 * x58)),
                  (x992 - (x993 * x61)),    (x994 - (x995 * x64)),    (x996 - (x997 * x67)),
                  (x998 - (x999 * x70)),    (x1000 - (x1001 * x73)),  (x1002 - (x1003 * x76)),
                  (x1004 - (x1005 * x79)),  (x1006 - (x1007 * x82)),  (x1008 - (x1009 * x85)),
                  (x1010 - (x1011 * x88)),  (x1012 - (x1013 * x91)),  (x1014 - (x1015 * x94)),
                  (x1016 - (x1017 * x97)),  (x1018 - (x1019 * x100)), (x1340 - (x1341 * x259)),
                  (x1342 - (x1343 * x262)), (x1344 - (x1345 * x265)), (x1346 - (x1347 * x268)),
                  (x1348 - (x1349 * x271)), (x1350 - (x1351 * x274)), (x1352 - (x1353 * x277)),
                  (x1354 - (x1355 * x280)), (x1356 - (x1357 * x283)), (x1358 - (x1359 * x286)),
                  (x1360 - (x1361 * x289)), (x1362 - (x1363 * x292)), (x1364 - (x1365 * x295)),
                  (x1366 - (x1367 * x298)), (x1368 - (x1369 * x301)), (x1370 - (x1371 * x304)),
                  (x1372 - (x1373 * x307)), (x1374 - (x1375 * x310)), (x1376 - (x1377 * x313)),
                  (x1378 - (x1379 * x316)), (x1380 - (x1381 * x319)), (x1382 - (x1383 * x322)),
                  (x1384 - (x1385 * x325)), (x1386 - (x1387 * x328)), (x1388 - (x1389 * x331)),
                  (x1390 - (x1391 * x334)), (x1392 - (x1393 * x337)), (x1394 - (x1395 * x340)),
                  (x1396 - (x1397 * x343)), (x1398 - (x1399 * x346)), (x1400 - (x1401 * x349)),
                  (x1402 - (x1403 * x352)), (x1724 - (x1725 * x448)), (x1726 - (x1727 * x451)),
                  (x1728 - (x1729 * x454)), (x1730 - (x1731 * x457)), (x1732 - (x1733 * x460)),
                  (x1734 - (x1735 * x463)), (x1736 - (x1737 * x466)), (x1738 - (x1739 * x469)),
                  (x1740 - (x1741 * x472)), (x1742 - (x1743 * x475)), (x1744 - (x1745 * x478)),
                  (x1746 - (x1747 * x481)), (x1748 - (x1749 * x484)), (x1750 - (x1751 * x487)),
                  (x1752 - (x1753 * x490)), (x1754 - (x1755 * x493)), (x1756 - (x1757 * x496)),
                  (x1758 - (x1759 * x499)), (x1760 - (x1761 * x502)), (x1762 - (x1763 * x505)),
                  (x1764 - (x1765 * x508)), (x1766 - (x1767 * x511)), (x1768 - (x1769 * x514)),
                  (x1770 - (x1771 * x517)), (x1772 - (x1773 * x520)), (x1774 - (x1775 * x523)),
                  (x1776 - (x1777 * x526)), (x1778 - (x1779 * x529)), (x1780 - (x1781 * x532)),
                  (x1782 - (x1783 * x535)), (x1784 - (x1785 * x538)), (x1786 - (x1787 * x541)),
                  (x2130 - (x2131 * x706)), (x2132 - (x2133 * x709)), (x2134 - (x2135 * x712)),
                  (x2136 - (x2137 * x715)), (x2138 - (x2139 * x718)), (x2140 - (x2141 * x721)),
                  (x2142 - (x2143 * x724)), (x2144 - (x2145 * x727)), (x2146 - (x2147 * x730)),
                  (x2148 - (x2149 * x733)), (x2150 - (x2151 * x736)), (x2152 - (x2153 * x739)),
                  (x2154 - (x2155 * x742)), (x2156 - (x2157 * x745)), (x2158 - (x2159 * x748)),
                  (x2160 - (x2161 * x751)), (x2162 - (x2163 * x754)), (x2164 - (x2165 * x757)),
                  (x2166 - (x2167 * x760)), (x2168 - (x2169 * x763)), (x2170 - (x2171 * x766)),
                  (x2108 - (x2109 * x577)), (x2110 - (x2111 * x580)), (x2112 - (x2113 * x583)),
                  (x2114 - (x2115 * x586)), (x2116 - (x2117 * x589)), (x2118 - (x2119 * x592)),
                  (x2120 - (x2121 * x595)), (x2122 - (x2123 * x598)), (x2124 - (x2125 * x601)),
                  (x2126 - (x2127 * x604)), (x2128 - (x2129 * x607)), (x2528 - (x2529 * x916)),
                  (x2530 - (x2531 * x919)), (x2532 - (x2533 * x922)), (x2534 - (x2535 * x925)),
                  (x2536 - (x2537 * x928)), (x2538 - (x2539 * x931)), (x2540 - (x2541 * x934)),
                  (x2542 - (x2543 * x937)), (x2544 - (x2545 * x940)), (x2546 - (x2547 * x943)),
                  (x2548 - (x2549 * x946)), (x2550 - (x2551 * x949)), (x2552 - (x2553 * x952)),
                  (x2554 - (x2555 * x955)), (x2492 - (x2493 * x769)), (x2494 - (x2495 * x772)),
                  (x2496 - (x2497 * x775)), (x2498 - (x2499 * x778)), (x2500 - (x2501 * x781)),
                  (x2502 - (x2503 * x784)), (x2504 - (x2505 * x787)), (x2506 - (x2507 * x790)),
                  (x2508 - (x2509 * x793)), (x2510 - (x2511 * x796)), (x2512 - (x2513 * x799)),
                  (x2514 - (x2515 * x802)), (x2516 - (x2517 * x805)), (x2518 - (x2519 * x808)),
                  (x2520 - (x2521 * x811)), (x2522 - (x2523 * x814)), (x2524 - (x2525 * x817)),
                  (x2526 - (x2527 * x820)), (x1156 - (x1157 * x685)), (x1158 - (x1159 * x688)),
                  (x1160 - (x1161 * x691)), (x1162 - (x1163 * x694)), (x1164 - (x1165 * x697)),
                  (x1166 - (x1167 * x700)), (x1168 - (x1169 * x703)), (x1170 - (x1171 * x706)),
                  (x1172 - (x1173 * x709)), (x1174 - (x1175 * x712)), (x1176 - (x1177 * x715)),
                  (x1178 - (x1179 * x718)), (x1180 - (x1181 * x721)), (x1182 - (x1183 * x724)),
                  (x1184 - (x1185 * x727)), (x1186 - (x1187 * x730)), (x1188 - (x1189 * x733)),
                  (x1190 - (x1191 * x736)), (x1192 - (x1193 * x739)), (x1194 - (x1195 * x742)),
                  (x1196 - (x1197 * x745)), (x1198 - (x1199 * x748)), (x1200 - (x1201 * x751)),
                  (x1202 - (x1203 * x754)), (x1204 - (x1205 * x757)), (x1206 - (x1207 * x760)),
                  (x1208 - (x1209 * x763)), (x1210 - (x1211 * x766)), (x1148 - (x1149 * x577)),
                  (x1150 - (x1151 * x580)), (x1152 - (x1153 * x583)), (x1154 - (x1155 * x586)),
                  (x1556 - (x1557 * x898)), (x1558 - (x1559 * x901)), (x1560 - (x1561 * x904)),
                  (x1562 - (x1563 * x907)), (x1564 - (x1565 * x910)), (x1566 - (x1567 * x913)),
                  (x1568 - (x1569 * x916)), (x1570 - (x1571 * x919)), (x1572 - (x1573 * x922)),
                  (x1574 - (x1575 * x925)), (x1576 - (x1577 * x928)), (x1578 - (x1579 * x931)),
                  (x1580 - (x1581 * x934)), (x1582 - (x1583 * x937)), (x1584 - (x1585 * x940)),
                  (x1586 - (x1587 * x943)), (x1588 - (x1589 * x946)), (x1590 - (x1591 * x949)),
                  (x1592 - (x1593 * x952)), (x1594 - (x1595 * x955)), (x1532 - (x1533 * x769)),
                  (x1534 - (x1535 * x772)), (x1536 - (x1537 * x775)), (x1538 - (x1539 * x778)),
                  (x1540 - (x1541 * x781)), (x1542 - (x1543 * x784)), (x1544 - (x1545 * x787)),
                  (x1546 - (x1547 * x790)), (x1548 - (x1549 * x793)), (x1550 - (x1551 * x796)),
                  (x1552 - (x1553 * x799)), (x1554 - (x1555 * x802)), (x1654 - (x1655 * x190)),
                  (x1656 - (x1657 * x193)), (x1658 - (x1659 * x196)), (x1596 - (x1597 * x7)),
                  (x1598 - (x1599 * x10)),  (x1600 - (x1601 * x13)),  (x1602 - (x1603 * x16)),
                  (x1604 - (x1605 * x19)),  (x1606 - (x1607 * x22)),  (x1608 - (x1609 * x25)),
                  (x1610 - (x1611 * x28)),  (x1612 - (x1613 * x31)),  (x1614 - (x1615 * x34)),
                  (x1616 - (x1617 * x37)),  (x1618 - (x1619 * x40)),  (x1620 - (x1621 * x43)),
                  (x1622 - (x1623 * x46)),  (x1624 - (x1625 * x49)),  (x1626 - (x1627 * x52)),
                  (x1628 - (x1629 * x55)),  (x1630 - (x1631 * x58)),  (x1632 - (x1633 * x61)),
                  (x1634 - (x1635 * x64)),  (x1636 - (x1637 * x67)),  (x1638 - (x1639 * x70)),
                  (x1640 - (x1641 * x73)),  (x1642 - (x1643 * x76)),  (x1644 - (x1645 * x79)),
                  (x1646 - (x1647 * x82)),  (x1648 - (x1649 * x85)),  (x1650 - (x1651 * x88)),
                  (x1652 - (x1653 * x91)),  (x1980 - (x1981 * x256)), (x1982 - (x1983 * x259)),
                  (x1984 - (x1985 * x262)), (x1986 - (x1987 * x265)), (x1988 - (x1989 * x268)),
                  (x1990 - (x1991 * x271)), (x1992 - (x1993 * x274)), (x1994 - (x1995 * x277)),
                  (x1996 - (x1997 * x280)), (x1998 - (x1999 * x283)), (x2000 - (x2001 * x286)),
                  (x2002 - (x2003 * x289)), (x2004 - (x2005 * x292)), (x2006 - (x2007 * x295)),
                  (x2008 - (x2009 * x298)), (x2010 - (x2011 * x301)), (x2012 - (x2013 * x304)),
                  (x2014 - (x2015 * x307)), (x2016 - (x2017 * x310)), (x2018 - (x2019 * x313)),
                  (x2020 - (x2021 * x316)), (x2022 - (x2023 * x319)), (x2024 - (x2025 * x322)),
                  (x2026 - (x2027 * x325)), (x2028 - (x2029 * x328)), (x2030 - (x2031 * x331)),
                  (x2032 - (x2033 * x334)), (x2034 - (x2035 * x337)), (x2036 - (x2037 * x340)),
                  (x2038 - (x2039 * x343)), (x2040 - (x2041 * x346)), (x2042 - (x2043 * x349)),
                  (x2364 - (x2365 * x394)), (x2366 - (x2367 * x397)), (x2368 - (x2369 * x400)),
                  (x2370 - (x2371 * x403)), (x2372 - (x2373 * x406)), (x2374 - (x2375 * x409)),
                  (x2376 - (x2377 * x412)), (x2378 - (x2379 * x415)), (x2380 - (x2381 * x418)),
                  (x2382 - (x2383 * x421)), (x2384 - (x2385 * x424)), (x2386 - (x2387 * x427)),
                  (x2388 - (x2389 * x430)), (x2390 - (x2391 * x433)), (x2392 - (x2393 * x436)),
                  (x2394 - (x2395 * x439)), (x2396 - (x2397 * x442)), (x2398 - (x2399 * x445)),
                  (x2400 - (x2401 * x448)), (x2402 - (x2403 * x451)), (x2404 - (x2405 * x454)),
                  (x2406 - (x2407 * x457)), (x2408 - (x2409 * x460)), (x2410 - (x2411 * x463)),
                  (x2412 - (x2413 * x466)), (x2414 - (x2415 * x469)), (x2416 - (x2417 * x472)),
                  (x2418 - (x2419 * x475)), (x2420 - (x2421 * x478)), (x2422 - (x2423 * x481)),
                  (x2424 - (x2425 * x484)), (x2426 - (x2427 * x487)), (x1082 - (x1083 * x382)),
                  (x1020 - (x1021 * x199)), (x1022 - (x1023 * x202)), (x1024 - (x1025 * x205)),
                  (x1026 - (x1027 * x208)), (x1028 - (x1029 * x211)), (x1030 - (x1031 * x214)),
                  (x1032 - (x1033 * x217)), (x1034 - (x1035 * x220)), (x1036 - (x1037 * x223)),
                  (x1038 - (x1039 * x226)), (x1040 - (x1041 * x229)), (x1042 - (x1043 * x232)),
                  (x1044 - (x1045 * x235)), (x1046 - (x1047 * x238)), (x1048 - (x1049 * x241)),
                  (x1050 - (x1051 * x244)), (x1052 - (x1053 * x247)), (x1054 - (x1055 * x250)),
                  (x1056 - (x1057 * x253)), (x1058 - (x1059 * x256)), (x1060 - (x1061 * x259)),
                  (x1062 - (x1063 * x262)), (x1064 - (x1065 * x265)), (x1066 - (x1067 * x268)),
                  (x1068 - (x1069 * x271)), (x1070 - (x1071 * x274)), (x1072 - (x1073 * x277)),
                  (x1074 - (x1075 * x280)), (x1076 - (x1077 * x283)), (x1078 - (x1079 * x286)),
                  (x1080 - (x1081 * x289)), (x1456 - (x1457 * x559)), (x1458 - (x1459 * x562)),
                  (x1460 - (x1461 * x565)), (x1462 - (x1463 * x568)), (x1464 - (x1465 * x571)),
                  (x1466 - (x1467 * x574)), (x1404 - (x1405 * x385)), (x1406 - (x1407 * x388)),
                  (x1408 - (x1409 * x391)), (x1410 - (x1411 * x394)), (x1412 - (x1413 * x397)),
                  (x1414 - (x1415 * x400)), (x1416 - (x1417 * x403)), (x1418 - (x1419 * x406)),
                  (x1420 - (x1421 * x409)), (x1422 - (x1423 * x412)), (x1424 - (x1425 * x415)),
                  (x1426 - (x1427 * x418)), (x1428 - (x1429 * x421)), (x1430 - (x1431 * x424)),
                  (x1432 - (x1433 * x427)), (x1434 - (x1435 * x430)), (x1436 - (x1437 * x433)),
                  (x1438 - (x1439 * x436)), (x1440 - (x1441 * x439)), (x1442 - (x1443 * x442)),
                  (x1444 - (x1445 * x445)), (x1446 - (x1447 * x448)), (x1448 - (x1449 * x451)),
                  (x1450 - (x1451 * x454)), (x1452 - (x1453 * x457)), (x1454 - (x1455 * x460)),
                  (x1802 - (x1803 * x694)), (x1804 - (x1805 * x697)), (x1806 - (x1807 * x700)),
                  (x1808 - (x1809 * x703)), (x1810 - (x1811 * x706)), (x1812 - (x1813 * x709)),
                  (x1814 - (x1815 * x712)), (x1816 - (x1817 * x715)), (x1818 - (x1819 * x718)),
                  (x1820 - (x1821 * x721)), (x1822 - (x1823 * x724)), (x1824 - (x1825 * x727)),
                  (x1826 - (x1827 * x730)), (x1828 - (x1829 * x733)), (x1830 - (x1831 * x736)),
                  (x1832 - (x1833 * x739)), (x1834 - (x1835 * x742)), (x1836 - (x1837 * x745)),
                  (x1838 - (x1839 * x748)), (x1840 - (x1841 * x751)), (x1842 - (x1843 * x754)),
                  (x1844 - (x1845 * x757)), (x1846 - (x1847 * x760)), (x1848 - (x1849 * x763)),
                  (x1850 - (x1851 * x766)), (x1788 - (x1789 * x577)), (x1790 - (x1791 * x580)),
                  (x1792 - (x1793 * x583)), (x1794 - (x1795 * x586)), (x1796 - (x1797 * x589)),
                  (x1798 - (x1799 * x592)), (x1800 - (x1801 * x595)), (x2220 - (x2221 * x934)),
                  (x2222 - (x2223 * x937)), (x2224 - (x2225 * x940)), (x2226 - (x2227 * x943)),
                  (x2228 - (x2229 * x946)), (x2230 - (x2231 * x949)), (x2232 - (x2233 * x952)),
                  (x2234 - (x2235 * x955)), (x2172 - (x2173 * x769)), (x2174 - (x2175 * x772)),
                  (x2176 - (x2177 * x775)), (x2178 - (x2179 * x778)), (x2180 - (x2181 * x781)),
                  (x2182 - (x2183 * x784)), (x2184 - (x2185 * x787)), (x2186 - (x2187 * x790)),
                  (x2188 - (x2189 * x793)), (x2190 - (x2191 * x796)), (x2192 - (x2193 * x799)),
                  (x2194 - (x2195 * x802)), (x2196 - (x2197 * x805)), (x2198 - (x2199 * x808)),
                  (x2200 - (x2201 * x811)), (x2202 - (x2203 * x814)), (x2204 - (x2205 * x817)),
                  (x2206 - (x2207 * x820)), (x2208 - (x2209 * x823)), (x2210 - (x2211 * x826)),
                  (x2212 - (x2213 * x829)), (x2214 - (x2215 * x832)), (x2216 - (x2217 * x835)),
                  (x2218 - (x2219 * x838)), (x2264 - (x2265 * x145)), (x2266 - (x2267 * x148)),
                  (x2268 - (x2269 * x151)), (x2270 - (x2271 * x154)), (x2272 - (x2273 * x157)),
                  (x2274 - (x2275 * x160)), (x2276 - (x2277 * x163)), (x2278 - (x2279 * x166)),
                  (x2280 - (x2281 * x169)), (x2282 - (x2283 * x172)), (x2284 - (x2285 * x175)),
                  (x2286 - (x2287 * x178)), (x2288 - (x2289 * x181)), (x2290 - (x2291 * x184)),
                  (x2292 - (x2293 * x187)), (x2294 - (x2295 * x190)), (x2296 - (x2297 * x193)),
                  (x2298 - (x2299 * x196)), (x2236 - (x2237 * x7)),   (x2238 - (x2239 * x10)),
                  (x2240 - (x2241 * x13)),  (x2242 - (x2243 * x16)),  (x2244 - (x2245 * x19)),
                  (x2246 - (x2247 * x22)),  (x2248 - (x2249 * x25)),  (x2250 - (x2251 * x28)),
                  (x2252 - (x2253 * x31)),  (x2254 - (x2255 * x34)),  (x2256 - (x2257 * x37)),
                  (x2258 - (x2259 * x40)),  (x2260 - (x2261 * x43)),  (x2262 - (x2263 * x46)),
                  (x1222 - (x1223 * x877)), (x1224 - (x1225 * x880)), (x1226 - (x1227 * x883)),
                  (x1228 - (x1229 * x886)), (x1230 - (x1231 * x889)), (x1232 - (x1233 * x892)),
                  (x1234 - (x1235 * x895)), (x1236 - (x1237 * x898)), (x1238 - (x1239 * x901)),
                  (x1240 - (x1241 * x904)), (x1242 - (x1243 * x907)), (x1244 - (x1245 * x910)),
                  (x1246 - (x1247 * x913)), (x1248 - (x1249 * x916)), (x1250 - (x1251 * x919)),
                  (x1252 - (x1253 * x922)), (x1254 - (x1255 * x925)), (x1256 - (x1257 * x928)),
                  (x1258 - (x1259 * x931)), (x1260 - (x1261 * x934)), (x1262 - (x1263 * x937)),
                  (x1264 - (x1265 * x940)), (x1266 - (x1267 * x943)), (x1268 - (x1269 * x946)),
                  (x1270 - (x1271 * x949)), (x1272 - (x1273 * x952)), (x1274 - (x1275 * x955)),
                  (x1212 - (x1213 * x769)), (x1214 - (x1215 * x772)), (x1216 - (x1217 * x775)),
                  (x1218 - (x1219 * x778)), (x1220 - (x1221 * x781)), (x1276 - (x1277 * x91)),
                  (x1278 - (x1279 * x94)),  (x1280 - (x1281 * x97)),  (x1282 - (x1283 * x100)),
                  (x1284 - (x1285 * x103)), (x1286 - (x1287 * x106)), (x1288 - (x1289 * x109)),
                  (x1290 - (x1291 * x112)), (x1292 - (x1293 * x115)), (x1294 - (x1295 * x118)),
                  (x1296 - (x1297 * x121)), (x1298 - (x1299 * x124)), (x1300 - (x1301 * x127)),
                  (x1302 - (x1303 * x130)), (x1304 - (x1305 * x133)), (x1306 - (x1307 * x136)),
                  (x1308 - (x1309 * x139)), (x1310 - (x1311 * x142)), (x1312 - (x1313 * x145)),
                  (x1314 - (x1315 * x148)), (x1316 - (x1317 * x151)), (x1318 - (x1319 * x154)),
                  (x1320 - (x1321 * x157)), (x1322 - (x1323 * x160)), (x1324 - (x1325 * x163)),
                  (x1326 - (x1327 * x166)), (x1328 - (x1329 * x169)), (x1330 - (x1331 * x172)),
                  (x1332 - (x1333 * x175)), (x1334 - (x1335 * x178)), (x1336 - (x1337 * x181)),
                  (x1338 - (x1339 * x184)), (x1704 - (x1705 * x355)), (x1706 - (x1707 * x358)),
                  (x1708 - (x1709 * x361)), (x1710 - (x1711 * x364)), (x1712 - (x1713 * x367)),
                  (x1714 - (x1715 * x370)), (x1716 - (x1717 * x373)), (x1718 - (x1719 * x376)),
                  (x1720 - (x1721 * x379)), (x1722 - (x1723 * x382)), (x1660 - (x1661 * x199)),
                  (x1662 - (x1663 * x202)), (x1664 - (x1665 * x205)), (x1666 - (x1667 * x208)),
                  (x1668 - (x1669 * x211)), (x1670 - (x1671 * x214)), (x1672 - (x1673 * x217)),
                  (x1674 - (x1675 * x220)), (x1676 - (x1677 * x223)), (x1678 - (x1679 * x226)),
                  (x1680 - (x1681 * x229)), (x1682 - (x1683 * x232)), (x1684 - (x1685 * x235)),
                  (x1686 - (x1687 * x238)), (x1688 - (x1689 * x241)), (x1690 - (x1691 * x244)),
                  (x1692 - (x1693 * x247)), (x1694 - (x1695 * x250)), (x1696 - (x1697 * x253)),
                  (x1698 - (x1699 * x256)), (x1700 - (x1701 * x259)), (x1702 - (x1703 * x262)),
                  (x2078 - (x2079 * x532)), (x2080 - (x2081 * x535)), (x2082 - (x2083 * x538)),
                  (x2084 - (x2085 * x541)), (x2086 - (x2087 * x544)), (x2088 - (x2089 * x547)),
                  (x2090 - (x2091 * x550)), (x2092 - (x2093 * x553)), (x2094 - (x2095 * x556)),
                  (x2096 - (x2097 * x559)), (x2098 - (x2099 * x562)), (x2100 - (x2101 * x565)),
                  (x2102 - (x2103 * x568)), (x2104 - (x2105 * x571)), (x2106 - (x2107 * x574)),
                  (x2044 - (x2045 * x385)), (x2046 - (x2047 * x388)), (x2048 - (x2049 * x391)),
                  (x2050 - (x2051 * x394)), (x2052 - (x2053 * x397)), (x2054 - (x2055 * x400)),
                  (x2056 - (x2057 * x403)), (x2058 - (x2059 * x406)), (x2060 - (x2061 * x409)),
                  (x2062 - (x2063 * x412)), (x2064 - (x2065 * x415)), (x2066 - (x2067 * x418)),
                  (x2068 - (x2069 * x421)), (x2070 - (x2071 * x424)), (x2072 - (x2073 * x427)),
                  (x2074 - (x2075 * x430)), (x2076 - (x2077 * x433)), (x2428 - (x2429 * x601)),
                  (x2430 - (x2431 * x604)), (x2432 - (x2433 * x607)), (x2434 - (x2435 * x610)),
                  (x2436 - (x2437 * x613)), (x2438 - (x2439 * x616)), (x2440 - (x2441 * x619)),
                  (x2442 - (x2443 * x622)), (x2444 - (x2445 * x625)), (x2446 - (x2447 * x628)),
                  (x2448 - (x2449 * x631)), (x2450 - (x2451 * x634)), (x2452 - (x2453 * x637)),
                  (x2454 - (x2455 * x640)), (x2456 - (x2457 * x643)), (x2458 - (x2459 * x646)),
                  (x2460 - (x2461 * x649)), (x2462 - (x2463 * x652)), (x2464 - (x2465 * x655)),
                  (x2466 - (x2467 * x658)), (x2468 - (x2469 * x661)), (x2470 - (x2471 * x664)),
                  (x2472 - (x2473 * x667)), (x2474 - (x2475 * x670)), (x2476 - (x2477 * x673)),
                  (x2478 - (x2479 * x676)), (x2480 - (x2481 * x679)), (x2482 - (x2483 * x682)),
                  (x2484 - (x2485 * x685)), (x2486 - (x2487 * x688)), (x2488 - (x2489 * x691)),
                  (x2490 - (x2491 * x694)), (x1084 - (x1085 * x391)), (x1086 - (x1087 * x394)),
                  (x1088 - (x1089 * x397)), (x1090 - (x1091 * x400)), (x1092 - (x1093 * x403)),
                  (x1094 - (x1095 * x406)), (x1096 - (x1097 * x409)), (x1098 - (x1099 * x412)),
                  (x1100 - (x1101 * x415)), (x1102 - (x1103 * x418)), (x1104 - (x1105 * x421)),
                  (x1106 - (x1107 * x424)), (x1108 - (x1109 * x427)), (x1110 - (x1111 * x430)),
                  (x1112 - (x1113 * x433)), (x1114 - (x1115 * x436)), (x1116 - (x1117 * x439)),
                  (x1118 - (x1119 * x442)), (x1120 - (x1121 * x445)), (x1122 - (x1123 * x448)),
                  (x1124 - (x1125 * x451)), (x1126 - (x1127 * x454)), (x1128 - (x1129 * x457)),
                  (x1130 - (x1131 * x460)), (x1132 - (x1133 * x463)), (x1134 - (x1135 * x466)),
                  (x1136 - (x1137 * x469)), (x1138 - (x1139 * x472)), (x1140 - (x1141 * x475)),
                  (x1142 - (x1143 * x478)), (x1144 - (x1145 * x481)), (x1146 - (x1147 * x484)),
                  (x1468 - (x1469 * x604)), (x1470 - (x1471 * x607)), (x1472 - (x1473 * x610)),
                  (x1474 - (x1475 * x613)), (x1476 - (x1477 * x616)), (x1478 - (x1479 * x619)),
                  (x1480 - (x1481 * x622)), (x1482 - (x1483 * x625)), (x1484 - (x1485 * x628)),
                  (x1486 - (x1487 * x631)), (x1488 - (x1489 * x634)), (x1490 - (x1491 * x637)),
                  (x1492 - (x1493 * x640)), (x1494 - (x1495 * x643)), (x1496 - (x1497 * x646)),
                  (x1498 - (x1499 * x649)), (x1500 - (x1501 * x652)), (x1502 - (x1503 * x655)),
                  (x1504 - (x1505 * x658)), (x1506 - (x1507 * x661)), (x1508 - (x1509 * x664)),
                  (x1510 - (x1511 * x667)), (x1512 - (x1513 * x670)), (x1514 - (x1515 * x673)),
                  (x1516 - (x1517 * x676)), (x1518 - (x1519 * x679)), (x1520 - (x1521 * x682)),
                  (x1522 - (x1523 * x685)), (x1524 - (x1525 * x688)), (x1526 - (x1527 * x691)),
                  (x1528 - (x1529 * x694)), (x1530 - (x1531 * x697)), (x1852 - (x1853 * x841)),
                  (x1854 - (x1855 * x844)), (x1856 - (x1857 * x847)), (x1858 - (x1859 * x850)),
                  (x1860 - (x1861 * x853)), (x1862 - (x1863 * x856)), (x1864 - (x1865 * x859)),
                  (x1866 - (x1867 * x862)), (x1868 - (x1869 * x865)), (x1870 - (x1871 * x868)),
                  (x1872 - (x1873 * x871)), (x1874 - (x1875 * x874)), (x1876 - (x1877 * x877)),
                  (x1878 - (x1879 * x880)), (x1880 - (x1881 * x883)), (x1882 - (x1883 * x886)),
                  (x1884 - (x1885 * x889)), (x1886 - (x1887 * x892)), (x1888 - (x1889 * x895)),
                  (x1890 - (x1891 * x898)), (x1892 - (x1893 * x901)), (x1894 - (x1895 * x904)),
                  (x1896 - (x1897 * x907)), (x1898 - (x1899 * x910)), (x1900 - (x1901 * x913)),
                  (x1902 - (x1903 * x916)), (x1904 - (x1905 * x919)), (x1906 - (x1907 * x922)),
                  (x1908 - (x1909 * x925)), (x1910 - (x1911 * x928)), (x1912 - (x1913 * x931)),
                  (x1914 - (x1915 * x934)), (x1916 - (x1917 * x76)),  (x1918 - (x1919 * x79)),
                  (x1920 - (x1921 * x82)),  (x1922 - (x1923 * x85)),  (x1924 - (x1925 * x88)),
                  (x1926 - (x1927 * x91)),  (x1928 - (x1929 * x94)),  (x1930 - (x1931 * x97)),
                  (x1932 - (x1933 * x100)), (x1934 - (x1935 * x103)), (x1936 - (x1937 * x106)),
                  (x1938 - (x1939 * x109)), (x1940 - (x1941 * x112)), (x1942 - (x1943 * x115)),
                  (x1944 - (x1945 * x118)), (x1946 - (x1947 * x121)), (x1948 - (x1949 * x124)),
                  (x1950 - (x1951 * x127)), (x1952 - (x1953 * x130)), (x1954 - (x1955 * x133)),
                  (x1956 - (x1957 * x136)), (x1958 - (x1959 * x139)), (x1960 - (x1961 * x142)),
                  (x1962 - (x1963 * x145)), (x1964 - (x1965 * x148)), (x1966 - (x1967 * x151)),
                  (x1968 - (x1969 * x154)), (x1970 - (x1971 * x157)), (x1972 - (x1973 * x160)),
                  (x1974 - (x1975 * x163)), (x1976 - (x1977 * x166)), (x1978 - (x1979 * x169)),
                  (x2360 - (x2361 * x379)), (x2362 - (x2363 * x382)), (x2300 - (x2301 * x199)),
                  (x2302 - (x2303 * x202)), (x2304 - (x2305 * x205)), (x2306 - (x2307 * x208)),
                  (x2308 - (x2309 * x211)), (x2310 - (x2311 * x214)), (x2312 - (x2313 * x217)),
                  (x2314 - (x2315 * x220)), (x2316 - (x2317 * x223)), (x2318 - (x2319 * x226)),
                  (x2320 - (x2321 * x229)), (x2322 - (x2323 * x232)), (x2324 - (x2325 * x235)),
                  (x2326 - (x2327 * x238)), (x2328 - (x2329 * x241)), (x2330 - (x2331 * x244)),
                  (x2332 - (x2333 * x247)), (x2334 - (x2335 * x250)), (x2336 - (x2337 * x253)),
                  (x2338 - (x2339 * x256)), (x2340 - (x2341 * x259)), (x2342 - (x2343 * x262)),
                  (x2344 - (x2345 * x265)), (x2346 - (x2347 * x268)), (x2348 - (x2349 * x271)),
                  (x2350 - (x2351 * x274)), (x2352 - (x2353 * x277)), (x2354 - (x2355 * x280)),
                  (x2356 - (x2357 * x283)), (x2358 - (x2359 * x286))},
      x2556,
      x2557,
      layout4);
  return x2558;
}

} // namespace risc0::circuit::keccak::cuda
