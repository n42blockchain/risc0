#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"

namespace risc0::impl {

__device__ TopStateStruct exec_ShaNextBlockCycle(ExecContext& ctx,
                                                 TopStateStruct arg0,
                                                 BoundLayout<ShaNextBlockCycleLayout> layout1) {
  // Log(<preamble>:22)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:236)
  INVOKE_EXTERN(ctx, log, "ShaNextBlockCycle", std::initializer_list<Val>{});
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:238)
  Val4Array x2 = Val4Array{Val(0), Val(1), Val(2), Val(3)};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:235)
  TopStateBitsStruct800Array x3 = arg0.bits;
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:238)
  Val x4 = x3[224]._super._super;
  Val x5 = x3[225]._super._super;
  Val x6 = x3[226]._super._super;
  Val x7 = x3[227]._super._super;
  Val x8 = x3[228]._super._super;
  Val x9 = x3[229]._super._super;
  Val x10 = x3[230]._super._super;
  Val x11 = x3[231]._super._super;
  Val x12 = x3[232]._super._super;
  Val x13 = x3[233]._super._super;
  Val x14 = x3[234]._super._super;
  Val x15 = x3[235]._super._super;
  Val x16 = x3[236]._super._super;
  Val x17 = x3[237]._super._super;
  Val x18 = x3[238]._super._super;
  Val x19 = x3[239]._super._super;
  Val x20 = x3[240]._super._super;
  Val x21 = x3[241]._super._super;
  Val x22 = x3[242]._super._super;
  Val x23 = x3[243]._super._super;
  Val x24 = x3[244]._super._super;
  Val x25 = x3[245]._super._super;
  Val x26 = x3[246]._super._super;
  Val x27 = x3[247]._super._super;
  Val x28 = x3[248]._super._super;
  Val x29 = x3[249]._super._super;
  Val x30 = x3[250]._super._super;
  Val x31 = x3[251]._super._super;
  Val x32 = x3[252]._super._super;
  Val x33 = x3[253]._super._super;
  Val x34 = x3[254]._super._super;
  Val x35 = x3[255]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x36 = (((x4 + (x5 * Val(2))) + (x6 * Val(4))) + (x7 * Val(8)));
  Val x37 = (((x36 + (x8 * Val(16))) + (x9 * Val(32))) + (x10 * Val(64)));
  Val x38 = (((x37 + (x11 * Val(128))) + (x12 * Val(256))) + (x13 * Val(512)));
  Val x39 = (((x38 + (x14 * Val(1024))) + (x15 * Val(2048))) + (x16 * Val(4096)));
  Val x40 = (((x39 + (x17 * Val(8192))) + (x18 * Val(16384))) + (x19 * Val(32768)));
  Val x41 = (((x20 + (x21 * Val(2))) + (x22 * Val(4))) + (x23 * Val(8)));
  Val x42 = (((x41 + (x24 * Val(16))) + (x25 * Val(32))) + (x26 * Val(64)));
  Val x43 = (((x42 + (x27 * Val(128))) + (x28 * Val(256))) + (x29 * Val(512)));
  Val x44 = (((x43 + (x30 * Val(1024))) + (x31 * Val(2048))) + (x32 * Val(4096)));
  Val x45 = (((x44 + (x33 * Val(8192))) + (x34 * Val(16384))) + (x35 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x46 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                  Pack_32__16__Super_SuperStruct{._super = x40},
                                                  Pack_32__16__Super_SuperStruct{._super = x45}}};
  Val x47 = x3[192]._super._super;
  Val x48 = x3[193]._super._super;
  Val x49 = x3[194]._super._super;
  Val x50 = x3[195]._super._super;
  Val x51 = x3[196]._super._super;
  Val x52 = x3[197]._super._super;
  Val x53 = x3[198]._super._super;
  Val x54 = x3[199]._super._super;
  Val x55 = x3[200]._super._super;
  Val x56 = x3[201]._super._super;
  Val x57 = x3[202]._super._super;
  Val x58 = x3[203]._super._super;
  Val x59 = x3[204]._super._super;
  Val x60 = x3[205]._super._super;
  Val x61 = x3[206]._super._super;
  Val x62 = x3[207]._super._super;
  Val x63 = x3[208]._super._super;
  Val x64 = x3[209]._super._super;
  Val x65 = x3[210]._super._super;
  Val x66 = x3[211]._super._super;
  Val x67 = x3[212]._super._super;
  Val x68 = x3[213]._super._super;
  Val x69 = x3[214]._super._super;
  Val x70 = x3[215]._super._super;
  Val x71 = x3[216]._super._super;
  Val x72 = x3[217]._super._super;
  Val x73 = x3[218]._super._super;
  Val x74 = x3[219]._super._super;
  Val x75 = x3[220]._super._super;
  Val x76 = x3[221]._super._super;
  Val x77 = x3[222]._super._super;
  Val x78 = x3[223]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x79 = (((x47 + (x48 * Val(2))) + (x49 * Val(4))) + (x50 * Val(8)));
  Val x80 = (((x79 + (x51 * Val(16))) + (x52 * Val(32))) + (x53 * Val(64)));
  Val x81 = (((x80 + (x54 * Val(128))) + (x55 * Val(256))) + (x56 * Val(512)));
  Val x82 = (((x81 + (x57 * Val(1024))) + (x58 * Val(2048))) + (x59 * Val(4096)));
  Val x83 = (((x82 + (x60 * Val(8192))) + (x61 * Val(16384))) + (x62 * Val(32768)));
  Val x84 = (((x63 + (x64 * Val(2))) + (x65 * Val(4))) + (x66 * Val(8)));
  Val x85 = (((x84 + (x67 * Val(16))) + (x68 * Val(32))) + (x69 * Val(64)));
  Val x86 = (((x85 + (x70 * Val(128))) + (x71 * Val(256))) + (x72 * Val(512)));
  Val x87 = (((x86 + (x73 * Val(1024))) + (x74 * Val(2048))) + (x75 * Val(4096)));
  Val x88 = (((x87 + (x76 * Val(8192))) + (x77 * Val(16384))) + (x78 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x89 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                  Pack_32__16__Super_SuperStruct{._super = x83},
                                                  Pack_32__16__Super_SuperStruct{._super = x88}}};
  Val x90 = x3[160]._super._super;
  Val x91 = x3[161]._super._super;
  Val x92 = x3[162]._super._super;
  Val x93 = x3[163]._super._super;
  Val x94 = x3[164]._super._super;
  Val x95 = x3[165]._super._super;
  Val x96 = x3[166]._super._super;
  Val x97 = x3[167]._super._super;
  Val x98 = x3[168]._super._super;
  Val x99 = x3[169]._super._super;
  Val x100 = x3[170]._super._super;
  Val x101 = x3[171]._super._super;
  Val x102 = x3[172]._super._super;
  Val x103 = x3[173]._super._super;
  Val x104 = x3[174]._super._super;
  Val x105 = x3[175]._super._super;
  Val x106 = x3[176]._super._super;
  Val x107 = x3[177]._super._super;
  Val x108 = x3[178]._super._super;
  Val x109 = x3[179]._super._super;
  Val x110 = x3[180]._super._super;
  Val x111 = x3[181]._super._super;
  Val x112 = x3[182]._super._super;
  Val x113 = x3[183]._super._super;
  Val x114 = x3[184]._super._super;
  Val x115 = x3[185]._super._super;
  Val x116 = x3[186]._super._super;
  Val x117 = x3[187]._super._super;
  Val x118 = x3[188]._super._super;
  Val x119 = x3[189]._super._super;
  Val x120 = x3[190]._super._super;
  Val x121 = x3[191]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x122 = (((x90 + (x91 * Val(2))) + (x92 * Val(4))) + (x93 * Val(8)));
  Val x123 = (((x122 + (x94 * Val(16))) + (x95 * Val(32))) + (x96 * Val(64)));
  Val x124 = (((x123 + (x97 * Val(128))) + (x98 * Val(256))) + (x99 * Val(512)));
  Val x125 = (((x124 + (x100 * Val(1024))) + (x101 * Val(2048))) + (x102 * Val(4096)));
  Val x126 = (((x125 + (x103 * Val(8192))) + (x104 * Val(16384))) + (x105 * Val(32768)));
  Val x127 = (((x106 + (x107 * Val(2))) + (x108 * Val(4))) + (x109 * Val(8)));
  Val x128 = (((x127 + (x110 * Val(16))) + (x111 * Val(32))) + (x112 * Val(64)));
  Val x129 = (((x128 + (x113 * Val(128))) + (x114 * Val(256))) + (x115 * Val(512)));
  Val x130 = (((x129 + (x116 * Val(1024))) + (x117 * Val(2048))) + (x118 * Val(4096)));
  Val x131 = (((x130 + (x119 * Val(8192))) + (x120 * Val(16384))) + (x121 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x132 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                   Pack_32__16__Super_SuperStruct{._super = x126},
                                                   Pack_32__16__Super_SuperStruct{._super = x131}}};
  Val x133 = x3[128]._super._super;
  Val x134 = x3[129]._super._super;
  Val x135 = x3[130]._super._super;
  Val x136 = x3[131]._super._super;
  Val x137 = x3[132]._super._super;
  Val x138 = x3[133]._super._super;
  Val x139 = x3[134]._super._super;
  Val x140 = x3[135]._super._super;
  Val x141 = x3[136]._super._super;
  Val x142 = x3[137]._super._super;
  Val x143 = x3[138]._super._super;
  Val x144 = x3[139]._super._super;
  Val x145 = x3[140]._super._super;
  Val x146 = x3[141]._super._super;
  Val x147 = x3[142]._super._super;
  Val x148 = x3[143]._super._super;
  Val x149 = x3[144]._super._super;
  Val x150 = x3[145]._super._super;
  Val x151 = x3[146]._super._super;
  Val x152 = x3[147]._super._super;
  Val x153 = x3[148]._super._super;
  Val x154 = x3[149]._super._super;
  Val x155 = x3[150]._super._super;
  Val x156 = x3[151]._super._super;
  Val x157 = x3[152]._super._super;
  Val x158 = x3[153]._super._super;
  Val x159 = x3[154]._super._super;
  Val x160 = x3[155]._super._super;
  Val x161 = x3[156]._super._super;
  Val x162 = x3[157]._super._super;
  Val x163 = x3[158]._super._super;
  Val x164 = x3[159]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x165 = (((x133 + (x134 * Val(2))) + (x135 * Val(4))) + (x136 * Val(8)));
  Val x166 = (((x165 + (x137 * Val(16))) + (x138 * Val(32))) + (x139 * Val(64)));
  Val x167 = (((x166 + (x140 * Val(128))) + (x141 * Val(256))) + (x142 * Val(512)));
  Val x168 = (((x167 + (x143 * Val(1024))) + (x144 * Val(2048))) + (x145 * Val(4096)));
  Val x169 = (((x168 + (x146 * Val(8192))) + (x147 * Val(16384))) + (x148 * Val(32768)));
  Val x170 = (((x149 + (x150 * Val(2))) + (x151 * Val(4))) + (x152 * Val(8)));
  Val x171 = (((x170 + (x153 * Val(16))) + (x154 * Val(32))) + (x155 * Val(64)));
  Val x172 = (((x171 + (x156 * Val(128))) + (x157 * Val(256))) + (x158 * Val(512)));
  Val x173 = (((x172 + (x159 * Val(1024))) + (x160 * Val(2048))) + (x161 * Val(4096)));
  Val x174 = (((x173 + (x162 * Val(8192))) + (x163 * Val(16384))) + (x164 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x175 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                   Pack_32__16__Super_SuperStruct{._super = x169},
                                                   Pack_32__16__Super_SuperStruct{._super = x174}}};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:239)
  Val x176 = x3[480]._super._super;
  Val x177 = x3[481]._super._super;
  Val x178 = x3[482]._super._super;
  Val x179 = x3[483]._super._super;
  Val x180 = x3[484]._super._super;
  Val x181 = x3[485]._super._super;
  Val x182 = x3[486]._super._super;
  Val x183 = x3[487]._super._super;
  Val x184 = x3[488]._super._super;
  Val x185 = x3[489]._super._super;
  Val x186 = x3[490]._super._super;
  Val x187 = x3[491]._super._super;
  Val x188 = x3[492]._super._super;
  Val x189 = x3[493]._super._super;
  Val x190 = x3[494]._super._super;
  Val x191 = x3[495]._super._super;
  Val x192 = x3[496]._super._super;
  Val x193 = x3[497]._super._super;
  Val x194 = x3[498]._super._super;
  Val x195 = x3[499]._super._super;
  Val x196 = x3[500]._super._super;
  Val x197 = x3[501]._super._super;
  Val x198 = x3[502]._super._super;
  Val x199 = x3[503]._super._super;
  Val x200 = x3[504]._super._super;
  Val x201 = x3[505]._super._super;
  Val x202 = x3[506]._super._super;
  Val x203 = x3[507]._super._super;
  Val x204 = x3[508]._super._super;
  Val x205 = x3[509]._super._super;
  Val x206 = x3[510]._super._super;
  Val x207 = x3[511]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x208 = (((x176 + (x177 * Val(2))) + (x178 * Val(4))) + (x179 * Val(8)));
  Val x209 = (((x208 + (x180 * Val(16))) + (x181 * Val(32))) + (x182 * Val(64)));
  Val x210 = (((x209 + (x183 * Val(128))) + (x184 * Val(256))) + (x185 * Val(512)));
  Val x211 = (((x210 + (x186 * Val(1024))) + (x187 * Val(2048))) + (x188 * Val(4096)));
  Val x212 = (((x211 + (x189 * Val(8192))) + (x190 * Val(16384))) + (x191 * Val(32768)));
  Val x213 = (((x192 + (x193 * Val(2))) + (x194 * Val(4))) + (x195 * Val(8)));
  Val x214 = (((x213 + (x196 * Val(16))) + (x197 * Val(32))) + (x198 * Val(64)));
  Val x215 = (((x214 + (x199 * Val(128))) + (x200 * Val(256))) + (x201 * Val(512)));
  Val x216 = (((x215 + (x202 * Val(1024))) + (x203 * Val(2048))) + (x204 * Val(4096)));
  Val x217 = (((x216 + (x205 * Val(8192))) + (x206 * Val(16384))) + (x207 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x218 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                   Pack_32__16__Super_SuperStruct{._super = x212},
                                                   Pack_32__16__Super_SuperStruct{._super = x217}}};
  Val x219 = x3[448]._super._super;
  Val x220 = x3[449]._super._super;
  Val x221 = x3[450]._super._super;
  Val x222 = x3[451]._super._super;
  Val x223 = x3[452]._super._super;
  Val x224 = x3[453]._super._super;
  Val x225 = x3[454]._super._super;
  Val x226 = x3[455]._super._super;
  Val x227 = x3[456]._super._super;
  Val x228 = x3[457]._super._super;
  Val x229 = x3[458]._super._super;
  Val x230 = x3[459]._super._super;
  Val x231 = x3[460]._super._super;
  Val x232 = x3[461]._super._super;
  Val x233 = x3[462]._super._super;
  Val x234 = x3[463]._super._super;
  Val x235 = x3[464]._super._super;
  Val x236 = x3[465]._super._super;
  Val x237 = x3[466]._super._super;
  Val x238 = x3[467]._super._super;
  Val x239 = x3[468]._super._super;
  Val x240 = x3[469]._super._super;
  Val x241 = x3[470]._super._super;
  Val x242 = x3[471]._super._super;
  Val x243 = x3[472]._super._super;
  Val x244 = x3[473]._super._super;
  Val x245 = x3[474]._super._super;
  Val x246 = x3[475]._super._super;
  Val x247 = x3[476]._super._super;
  Val x248 = x3[477]._super._super;
  Val x249 = x3[478]._super._super;
  Val x250 = x3[479]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x251 = (((x219 + (x220 * Val(2))) + (x221 * Val(4))) + (x222 * Val(8)));
  Val x252 = (((x251 + (x223 * Val(16))) + (x224 * Val(32))) + (x225 * Val(64)));
  Val x253 = (((x252 + (x226 * Val(128))) + (x227 * Val(256))) + (x228 * Val(512)));
  Val x254 = (((x253 + (x229 * Val(1024))) + (x230 * Val(2048))) + (x231 * Val(4096)));
  Val x255 = (((x254 + (x232 * Val(8192))) + (x233 * Val(16384))) + (x234 * Val(32768)));
  Val x256 = (((x235 + (x236 * Val(2))) + (x237 * Val(4))) + (x238 * Val(8)));
  Val x257 = (((x256 + (x239 * Val(16))) + (x240 * Val(32))) + (x241 * Val(64)));
  Val x258 = (((x257 + (x242 * Val(128))) + (x243 * Val(256))) + (x244 * Val(512)));
  Val x259 = (((x258 + (x245 * Val(1024))) + (x246 * Val(2048))) + (x247 * Val(4096)));
  Val x260 = (((x259 + (x248 * Val(8192))) + (x249 * Val(16384))) + (x250 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x261 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                   Pack_32__16__Super_SuperStruct{._super = x255},
                                                   Pack_32__16__Super_SuperStruct{._super = x260}}};
  Val x262 = x3[416]._super._super;
  Val x263 = x3[417]._super._super;
  Val x264 = x3[418]._super._super;
  Val x265 = x3[419]._super._super;
  Val x266 = x3[420]._super._super;
  Val x267 = x3[421]._super._super;
  Val x268 = x3[422]._super._super;
  Val x269 = x3[423]._super._super;
  Val x270 = x3[424]._super._super;
  Val x271 = x3[425]._super._super;
  Val x272 = x3[426]._super._super;
  Val x273 = x3[427]._super._super;
  Val x274 = x3[428]._super._super;
  Val x275 = x3[429]._super._super;
  Val x276 = x3[430]._super._super;
  Val x277 = x3[431]._super._super;
  Val x278 = x3[432]._super._super;
  Val x279 = x3[433]._super._super;
  Val x280 = x3[434]._super._super;
  Val x281 = x3[435]._super._super;
  Val x282 = x3[436]._super._super;
  Val x283 = x3[437]._super._super;
  Val x284 = x3[438]._super._super;
  Val x285 = x3[439]._super._super;
  Val x286 = x3[440]._super._super;
  Val x287 = x3[441]._super._super;
  Val x288 = x3[442]._super._super;
  Val x289 = x3[443]._super._super;
  Val x290 = x3[444]._super._super;
  Val x291 = x3[445]._super._super;
  Val x292 = x3[446]._super._super;
  Val x293 = x3[447]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x294 = (((x262 + (x263 * Val(2))) + (x264 * Val(4))) + (x265 * Val(8)));
  Val x295 = (((x294 + (x266 * Val(16))) + (x267 * Val(32))) + (x268 * Val(64)));
  Val x296 = (((x295 + (x269 * Val(128))) + (x270 * Val(256))) + (x271 * Val(512)));
  Val x297 = (((x296 + (x272 * Val(1024))) + (x273 * Val(2048))) + (x274 * Val(4096)));
  Val x298 = (((x297 + (x275 * Val(8192))) + (x276 * Val(16384))) + (x277 * Val(32768)));
  Val x299 = (((x278 + (x279 * Val(2))) + (x280 * Val(4))) + (x281 * Val(8)));
  Val x300 = (((x299 + (x282 * Val(16))) + (x283 * Val(32))) + (x284 * Val(64)));
  Val x301 = (((x300 + (x285 * Val(128))) + (x286 * Val(256))) + (x287 * Val(512)));
  Val x302 = (((x301 + (x288 * Val(1024))) + (x289 * Val(2048))) + (x290 * Val(4096)));
  Val x303 = (((x302 + (x291 * Val(8192))) + (x292 * Val(16384))) + (x293 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x304 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                   Pack_32__16__Super_SuperStruct{._super = x298},
                                                   Pack_32__16__Super_SuperStruct{._super = x303}}};
  Val x305 = x3[384]._super._super;
  Val x306 = x3[385]._super._super;
  Val x307 = x3[386]._super._super;
  Val x308 = x3[387]._super._super;
  Val x309 = x3[388]._super._super;
  Val x310 = x3[389]._super._super;
  Val x311 = x3[390]._super._super;
  Val x312 = x3[391]._super._super;
  Val x313 = x3[392]._super._super;
  Val x314 = x3[393]._super._super;
  Val x315 = x3[394]._super._super;
  Val x316 = x3[395]._super._super;
  Val x317 = x3[396]._super._super;
  Val x318 = x3[397]._super._super;
  Val x319 = x3[398]._super._super;
  Val x320 = x3[399]._super._super;
  Val x321 = x3[400]._super._super;
  Val x322 = x3[401]._super._super;
  Val x323 = x3[402]._super._super;
  Val x324 = x3[403]._super._super;
  Val x325 = x3[404]._super._super;
  Val x326 = x3[405]._super._super;
  Val x327 = x3[406]._super._super;
  Val x328 = x3[407]._super._super;
  Val x329 = x3[408]._super._super;
  Val x330 = x3[409]._super._super;
  Val x331 = x3[410]._super._super;
  Val x332 = x3[411]._super._super;
  Val x333 = x3[412]._super._super;
  Val x334 = x3[413]._super._super;
  Val x335 = x3[414]._super._super;
  Val x336 = x3[415]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x337 = (((x305 + (x306 * Val(2))) + (x307 * Val(4))) + (x308 * Val(8)));
  Val x338 = (((x337 + (x309 * Val(16))) + (x310 * Val(32))) + (x311 * Val(64)));
  Val x339 = (((x338 + (x312 * Val(128))) + (x313 * Val(256))) + (x314 * Val(512)));
  Val x340 = (((x339 + (x315 * Val(1024))) + (x316 * Val(2048))) + (x317 * Val(4096)));
  Val x341 = (((x340 + (x318 * Val(8192))) + (x319 * Val(16384))) + (x320 * Val(32768)));
  Val x342 = (((x321 + (x322 * Val(2))) + (x323 * Val(4))) + (x324 * Val(8)));
  Val x343 = (((x342 + (x325 * Val(16))) + (x326 * Val(32))) + (x327 * Val(64)));
  Val x344 = (((x343 + (x328 * Val(128))) + (x329 * Val(256))) + (x330 * Val(512)));
  Val x345 = (((x344 + (x331 * Val(1024))) + (x332 * Val(2048))) + (x333 * Val(4096)));
  Val x346 = (((x345 + (x334 * Val(8192))) + (x335 * Val(16384))) + (x336 * Val(32768)));
  // Pack(zirgen/circuit/keccak2/pack.zir:29)
  Pack_32__16_Struct x347 = Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{
                                                   Pack_32__16__Super_SuperStruct{._super = x341},
                                                   Pack_32__16__Super_SuperStruct{._super = x346}}};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:235)
  NondetRegStruct16Array x348 = arg0.sflat;
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:241)
  ShaNextBlockCyclePrevA_SuperStruct x349 =
      ShaNextBlockCyclePrevA_SuperStruct{._super = NondetRegStruct2Array{x348[0], x348[1]}};
  ShaNextBlockCyclePrevA_SuperStruct x350 =
      ShaNextBlockCyclePrevA_SuperStruct{._super = NondetRegStruct2Array{x348[2], x348[3]}};
  ShaNextBlockCyclePrevA_SuperStruct x351 =
      ShaNextBlockCyclePrevA_SuperStruct{._super = NondetRegStruct2Array{x348[4], x348[5]}};
  ShaNextBlockCyclePrevA_SuperStruct x352 =
      ShaNextBlockCyclePrevA_SuperStruct{._super = NondetRegStruct2Array{x348[6], x348[7]}};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:242)
  ShaNextBlockCyclePrevE_SuperStruct x353 =
      ShaNextBlockCyclePrevE_SuperStruct{._super = NondetRegStruct2Array{x348[8], x348[9]}};
  ShaNextBlockCyclePrevE_SuperStruct x354 =
      ShaNextBlockCyclePrevE_SuperStruct{._super = NondetRegStruct2Array{x348[10], x348[11]}};
  ShaNextBlockCyclePrevE_SuperStruct x355 =
      ShaNextBlockCyclePrevE_SuperStruct{._super = NondetRegStruct2Array{x348[12], x348[13]}};
  ShaNextBlockCyclePrevE_SuperStruct x356 =
      ShaNextBlockCyclePrevE_SuperStruct{._super = NondetRegStruct2Array{x348[14], x348[15]}};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:244)
  UnpackReg_32__16_Struct4Array x357 =
      map(x2,
          LAYOUT_LOOKUP(layout1, totA),
          ([&](Val4Array::value_type x358,
               BoundLayout<CarryAndExpandLayout4LayoutArray::value_type> x359) {
            // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:238)
            Pack_32__16__Super_SuperStruct2Array x360 =
                Pack_32__16_Struct4Array{x46, x89, x132, x175}[to_size_t(x358)]._super;
            // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:241)
            NondetRegStruct2Array x361 =
                ShaNextBlockCyclePrevA_SuperStruct4Array{x349, x350, x351, x352}[to_size_t(x358)]
                    ._super;
            // Add2(zirgen/circuit/keccak2/sha2.zir:30)
            // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:244)
            Val x362 = (x360[0]._super + x361[0]._super);
            Val x363 = (x360[1]._super + x361[1]._super);
            UnpackReg_32__16_Struct x364 = exec_CarryAndExpand(ctx, Val2Array{x362, x363}, x359);
            return x364;
          }));
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:245)
  UnpackReg_32__16_Struct4Array x365 =
      map(x2,
          LAYOUT_LOOKUP(layout1, totE),
          ([&](Val4Array::value_type x366,
               BoundLayout<CarryAndExpandLayout4LayoutArray::value_type> x367) {
            // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:239)
            Pack_32__16__Super_SuperStruct2Array x368 =
                Pack_32__16_Struct4Array{x218, x261, x304, x347}[to_size_t(x366)]._super;
            // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:242)
            NondetRegStruct2Array x369 =
                ShaNextBlockCyclePrevE_SuperStruct4Array{x353, x354, x355, x356}[to_size_t(x366)]
                    ._super;
            // Add2(zirgen/circuit/keccak2/sha2.zir:30)
            // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:245)
            Val x370 = (x368[0]._super + x369[0]._super);
            Val x371 = (x368[1]._super + x369[1]._super);
            UnpackReg_32__16_Struct x372 = exec_CarryAndExpand(ctx, Val2Array{x370, x371}, x367);
            return x372;
          }));
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:244)
  UnpackReg_32__16__SuperStruct32Array x373 = x357[3]._super;
  Val x374 = x373[0]._super._super;
  Val x375 = x373[1]._super._super;
  Val x376 = x373[2]._super._super;
  Val x377 = x373[3]._super._super;
  Val x378 = x373[4]._super._super;
  Val x379 = x373[5]._super._super;
  Val x380 = x373[6]._super._super;
  Val x381 = x373[7]._super._super;
  Val x382 = x373[8]._super._super;
  Val x383 = x373[9]._super._super;
  Val x384 = x373[10]._super._super;
  Val x385 = x373[11]._super._super;
  Val x386 = x373[12]._super._super;
  Val x387 = x373[13]._super._super;
  Val x388 = x373[14]._super._super;
  Val x389 = x373[15]._super._super;
  Val x390 = x373[16]._super._super;
  Val x391 = x373[17]._super._super;
  Val x392 = x373[18]._super._super;
  Val x393 = x373[19]._super._super;
  Val x394 = x373[20]._super._super;
  Val x395 = x373[21]._super._super;
  Val x396 = x373[22]._super._super;
  Val x397 = x373[23]._super._super;
  Val x398 = x373[24]._super._super;
  Val x399 = x373[25]._super._super;
  Val x400 = x373[26]._super._super;
  Val x401 = x373[27]._super._super;
  Val x402 = x373[28]._super._super;
  Val x403 = x373[29]._super._super;
  Val x404 = x373[30]._super._super;
  Val x405 = x373[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x406 = x357[2]._super;
  Val x407 = x406[0]._super._super;
  Val x408 = x406[1]._super._super;
  Val x409 = x406[2]._super._super;
  Val x410 = x406[3]._super._super;
  Val x411 = x406[4]._super._super;
  Val x412 = x406[5]._super._super;
  Val x413 = x406[6]._super._super;
  Val x414 = x406[7]._super._super;
  Val x415 = x406[8]._super._super;
  Val x416 = x406[9]._super._super;
  Val x417 = x406[10]._super._super;
  Val x418 = x406[11]._super._super;
  Val x419 = x406[12]._super._super;
  Val x420 = x406[13]._super._super;
  Val x421 = x406[14]._super._super;
  Val x422 = x406[15]._super._super;
  Val x423 = x406[16]._super._super;
  Val x424 = x406[17]._super._super;
  Val x425 = x406[18]._super._super;
  Val x426 = x406[19]._super._super;
  Val x427 = x406[20]._super._super;
  Val x428 = x406[21]._super._super;
  Val x429 = x406[22]._super._super;
  Val x430 = x406[23]._super._super;
  Val x431 = x406[24]._super._super;
  Val x432 = x406[25]._super._super;
  Val x433 = x406[26]._super._super;
  Val x434 = x406[27]._super._super;
  Val x435 = x406[28]._super._super;
  Val x436 = x406[29]._super._super;
  Val x437 = x406[30]._super._super;
  Val x438 = x406[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x439 = x357[1]._super;
  Val x440 = x439[0]._super._super;
  Val x441 = x439[1]._super._super;
  Val x442 = x439[2]._super._super;
  Val x443 = x439[3]._super._super;
  Val x444 = x439[4]._super._super;
  Val x445 = x439[5]._super._super;
  Val x446 = x439[6]._super._super;
  Val x447 = x439[7]._super._super;
  Val x448 = x439[8]._super._super;
  Val x449 = x439[9]._super._super;
  Val x450 = x439[10]._super._super;
  Val x451 = x439[11]._super._super;
  Val x452 = x439[12]._super._super;
  Val x453 = x439[13]._super._super;
  Val x454 = x439[14]._super._super;
  Val x455 = x439[15]._super._super;
  Val x456 = x439[16]._super._super;
  Val x457 = x439[17]._super._super;
  Val x458 = x439[18]._super._super;
  Val x459 = x439[19]._super._super;
  Val x460 = x439[20]._super._super;
  Val x461 = x439[21]._super._super;
  Val x462 = x439[22]._super._super;
  Val x463 = x439[23]._super._super;
  Val x464 = x439[24]._super._super;
  Val x465 = x439[25]._super._super;
  Val x466 = x439[26]._super._super;
  Val x467 = x439[27]._super._super;
  Val x468 = x439[28]._super._super;
  Val x469 = x439[29]._super._super;
  Val x470 = x439[30]._super._super;
  Val x471 = x439[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x472 = x357[0]._super;
  Val x473 = x472[0]._super._super;
  Val x474 = x472[1]._super._super;
  Val x475 = x472[2]._super._super;
  Val x476 = x472[3]._super._super;
  Val x477 = x472[4]._super._super;
  Val x478 = x472[5]._super._super;
  Val x479 = x472[6]._super._super;
  Val x480 = x472[7]._super._super;
  Val x481 = x472[8]._super._super;
  Val x482 = x472[9]._super._super;
  Val x483 = x472[10]._super._super;
  Val x484 = x472[11]._super._super;
  Val x485 = x472[12]._super._super;
  Val x486 = x472[13]._super._super;
  Val x487 = x472[14]._super._super;
  Val x488 = x472[15]._super._super;
  Val x489 = x472[16]._super._super;
  Val x490 = x472[17]._super._super;
  Val x491 = x472[18]._super._super;
  Val x492 = x472[19]._super._super;
  Val x493 = x472[20]._super._super;
  Val x494 = x472[21]._super._super;
  Val x495 = x472[22]._super._super;
  Val x496 = x472[23]._super._super;
  Val x497 = x472[24]._super._super;
  Val x498 = x472[25]._super._super;
  Val x499 = x472[26]._super._super;
  Val x500 = x472[27]._super._super;
  Val x501 = x472[28]._super._super;
  Val x502 = x472[29]._super._super;
  Val x503 = x472[30]._super._super;
  Val x504 = x472[31]._super._super;
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:245)
  UnpackReg_32__16__SuperStruct32Array x505 = x365[3]._super;
  Val x506 = x505[0]._super._super;
  Val x507 = x505[1]._super._super;
  Val x508 = x505[2]._super._super;
  Val x509 = x505[3]._super._super;
  Val x510 = x505[4]._super._super;
  Val x511 = x505[5]._super._super;
  Val x512 = x505[6]._super._super;
  Val x513 = x505[7]._super._super;
  Val x514 = x505[8]._super._super;
  Val x515 = x505[9]._super._super;
  Val x516 = x505[10]._super._super;
  Val x517 = x505[11]._super._super;
  Val x518 = x505[12]._super._super;
  Val x519 = x505[13]._super._super;
  Val x520 = x505[14]._super._super;
  Val x521 = x505[15]._super._super;
  Val x522 = x505[16]._super._super;
  Val x523 = x505[17]._super._super;
  Val x524 = x505[18]._super._super;
  Val x525 = x505[19]._super._super;
  Val x526 = x505[20]._super._super;
  Val x527 = x505[21]._super._super;
  Val x528 = x505[22]._super._super;
  Val x529 = x505[23]._super._super;
  Val x530 = x505[24]._super._super;
  Val x531 = x505[25]._super._super;
  Val x532 = x505[26]._super._super;
  Val x533 = x505[27]._super._super;
  Val x534 = x505[28]._super._super;
  Val x535 = x505[29]._super._super;
  Val x536 = x505[30]._super._super;
  Val x537 = x505[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x538 = x365[2]._super;
  Val x539 = x538[0]._super._super;
  Val x540 = x538[1]._super._super;
  Val x541 = x538[2]._super._super;
  Val x542 = x538[3]._super._super;
  Val x543 = x538[4]._super._super;
  Val x544 = x538[5]._super._super;
  Val x545 = x538[6]._super._super;
  Val x546 = x538[7]._super._super;
  Val x547 = x538[8]._super._super;
  Val x548 = x538[9]._super._super;
  Val x549 = x538[10]._super._super;
  Val x550 = x538[11]._super._super;
  Val x551 = x538[12]._super._super;
  Val x552 = x538[13]._super._super;
  Val x553 = x538[14]._super._super;
  Val x554 = x538[15]._super._super;
  Val x555 = x538[16]._super._super;
  Val x556 = x538[17]._super._super;
  Val x557 = x538[18]._super._super;
  Val x558 = x538[19]._super._super;
  Val x559 = x538[20]._super._super;
  Val x560 = x538[21]._super._super;
  Val x561 = x538[22]._super._super;
  Val x562 = x538[23]._super._super;
  Val x563 = x538[24]._super._super;
  Val x564 = x538[25]._super._super;
  Val x565 = x538[26]._super._super;
  Val x566 = x538[27]._super._super;
  Val x567 = x538[28]._super._super;
  Val x568 = x538[29]._super._super;
  Val x569 = x538[30]._super._super;
  Val x570 = x538[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x571 = x365[1]._super;
  Val x572 = x571[0]._super._super;
  Val x573 = x571[1]._super._super;
  Val x574 = x571[2]._super._super;
  Val x575 = x571[3]._super._super;
  Val x576 = x571[4]._super._super;
  Val x577 = x571[5]._super._super;
  Val x578 = x571[6]._super._super;
  Val x579 = x571[7]._super._super;
  Val x580 = x571[8]._super._super;
  Val x581 = x571[9]._super._super;
  Val x582 = x571[10]._super._super;
  Val x583 = x571[11]._super._super;
  Val x584 = x571[12]._super._super;
  Val x585 = x571[13]._super._super;
  Val x586 = x571[14]._super._super;
  Val x587 = x571[15]._super._super;
  Val x588 = x571[16]._super._super;
  Val x589 = x571[17]._super._super;
  Val x590 = x571[18]._super._super;
  Val x591 = x571[19]._super._super;
  Val x592 = x571[20]._super._super;
  Val x593 = x571[21]._super._super;
  Val x594 = x571[22]._super._super;
  Val x595 = x571[23]._super._super;
  Val x596 = x571[24]._super._super;
  Val x597 = x571[25]._super._super;
  Val x598 = x571[26]._super._super;
  Val x599 = x571[27]._super._super;
  Val x600 = x571[28]._super._super;
  Val x601 = x571[29]._super._super;
  Val x602 = x571[30]._super._super;
  Val x603 = x571[31]._super._super;
  UnpackReg_32__16__SuperStruct32Array x604 = x365[0]._super;
  Val x605 = x604[0]._super._super;
  Val x606 = x604[1]._super._super;
  Val x607 = x604[2]._super._super;
  Val x608 = x604[3]._super._super;
  Val x609 = x604[4]._super._super;
  Val x610 = x604[5]._super._super;
  Val x611 = x604[6]._super._super;
  Val x612 = x604[7]._super._super;
  Val x613 = x604[8]._super._super;
  Val x614 = x604[9]._super._super;
  Val x615 = x604[10]._super._super;
  Val x616 = x604[11]._super._super;
  Val x617 = x604[12]._super._super;
  Val x618 = x604[13]._super._super;
  Val x619 = x604[14]._super._super;
  Val x620 = x604[15]._super._super;
  Val x621 = x604[16]._super._super;
  Val x622 = x604[17]._super._super;
  Val x623 = x604[18]._super._super;
  Val x624 = x604[19]._super._super;
  Val x625 = x604[20]._super._super;
  Val x626 = x604[21]._super._super;
  Val x627 = x604[22]._super._super;
  Val x628 = x604[23]._super._super;
  Val x629 = x604[24]._super._super;
  Val x630 = x604[25]._super._super;
  Val x631 = x604[26]._super._super;
  Val x632 = x604[27]._super._super;
  Val x633 = x604[28]._super._super;
  Val x634 = x604[29]._super._super;
  Val x635 = x604[30]._super._super;
  Val x636 = x604[31]._super._super;
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:257)
  Val x637 = (((x473 + (x474 * Val(2))) + (x475 * Val(4))) + (x476 * Val(8)));
  Val x638 = (((x637 + (x477 * Val(16))) + (x478 * Val(32))) + (x479 * Val(64)));
  Val x639 = (((x638 + (x480 * Val(128))) + (x481 * Val(256))) + (x482 * Val(512)));
  Val x640 = (((x639 + (x483 * Val(1024))) + (x484 * Val(2048))) + (x485 * Val(4096)));
  Val x641 = (((x640 + (x486 * Val(8192))) + (x487 * Val(16384))) + (x488 * Val(32768)));
  Val x642 = (((x489 + (x490 * Val(2))) + (x491 * Val(4))) + (x492 * Val(8)));
  Val x643 = (((x642 + (x493 * Val(16))) + (x494 * Val(32))) + (x495 * Val(64)));
  Val x644 = (((x643 + (x496 * Val(128))) + (x497 * Val(256))) + (x498 * Val(512)));
  Val x645 = (((x644 + (x499 * Val(1024))) + (x500 * Val(2048))) + (x501 * Val(4096)));
  Val x646 = (((x645 + (x502 * Val(8192))) + (x503 * Val(16384))) + (x504 * Val(32768)));
  Val x647 = (((x440 + (x441 * Val(2))) + (x442 * Val(4))) + (x443 * Val(8)));
  Val x648 = (((x647 + (x444 * Val(16))) + (x445 * Val(32))) + (x446 * Val(64)));
  Val x649 = (((x648 + (x447 * Val(128))) + (x448 * Val(256))) + (x449 * Val(512)));
  Val x650 = (((x649 + (x450 * Val(1024))) + (x451 * Val(2048))) + (x452 * Val(4096)));
  Val x651 = (((x650 + (x453 * Val(8192))) + (x454 * Val(16384))) + (x455 * Val(32768)));
  Val x652 = (((x456 + (x457 * Val(2))) + (x458 * Val(4))) + (x459 * Val(8)));
  Val x653 = (((x652 + (x460 * Val(16))) + (x461 * Val(32))) + (x462 * Val(64)));
  Val x654 = (((x653 + (x463 * Val(128))) + (x464 * Val(256))) + (x465 * Val(512)));
  Val x655 = (((x654 + (x466 * Val(1024))) + (x467 * Val(2048))) + (x468 * Val(4096)));
  Val x656 = (((x655 + (x469 * Val(8192))) + (x470 * Val(16384))) + (x471 * Val(32768)));
  Val x657 = (((x407 + (x408 * Val(2))) + (x409 * Val(4))) + (x410 * Val(8)));
  Val x658 = (((x657 + (x411 * Val(16))) + (x412 * Val(32))) + (x413 * Val(64)));
  Val x659 = (((x658 + (x414 * Val(128))) + (x415 * Val(256))) + (x416 * Val(512)));
  Val x660 = (((x659 + (x417 * Val(1024))) + (x418 * Val(2048))) + (x419 * Val(4096)));
  Val x661 = (((x660 + (x420 * Val(8192))) + (x421 * Val(16384))) + (x422 * Val(32768)));
  Val x662 = (((x423 + (x424 * Val(2))) + (x425 * Val(4))) + (x426 * Val(8)));
  Val x663 = (((x662 + (x427 * Val(16))) + (x428 * Val(32))) + (x429 * Val(64)));
  Val x664 = (((x663 + (x430 * Val(128))) + (x431 * Val(256))) + (x432 * Val(512)));
  Val x665 = (((x664 + (x433 * Val(1024))) + (x434 * Val(2048))) + (x435 * Val(4096)));
  Val x666 = (((x665 + (x436 * Val(8192))) + (x437 * Val(16384))) + (x438 * Val(32768)));
  Val x667 = (((x374 + (x375 * Val(2))) + (x376 * Val(4))) + (x377 * Val(8)));
  Val x668 = (((x667 + (x378 * Val(16))) + (x379 * Val(32))) + (x380 * Val(64)));
  Val x669 = (((x668 + (x381 * Val(128))) + (x382 * Val(256))) + (x383 * Val(512)));
  Val x670 = (((x669 + (x384 * Val(1024))) + (x385 * Val(2048))) + (x386 * Val(4096)));
  Val x671 = (((x670 + (x387 * Val(8192))) + (x388 * Val(16384))) + (x389 * Val(32768)));
  Val x672 = (((x390 + (x391 * Val(2))) + (x392 * Val(4))) + (x393 * Val(8)));
  Val x673 = (((x672 + (x394 * Val(16))) + (x395 * Val(32))) + (x396 * Val(64)));
  Val x674 = (((x673 + (x397 * Val(128))) + (x398 * Val(256))) + (x399 * Val(512)));
  Val x675 = (((x674 + (x400 * Val(1024))) + (x401 * Val(2048))) + (x402 * Val(4096)));
  Val x676 = (((x675 + (x403 * Val(8192))) + (x404 * Val(16384))) + (x405 * Val(32768)));
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:259)
  Val x677 = (((x605 + (x606 * Val(2))) + (x607 * Val(4))) + (x608 * Val(8)));
  Val x678 = (((x677 + (x609 * Val(16))) + (x610 * Val(32))) + (x611 * Val(64)));
  Val x679 = (((x678 + (x612 * Val(128))) + (x613 * Val(256))) + (x614 * Val(512)));
  Val x680 = (((x679 + (x615 * Val(1024))) + (x616 * Val(2048))) + (x617 * Val(4096)));
  Val x681 = (((x680 + (x618 * Val(8192))) + (x619 * Val(16384))) + (x620 * Val(32768)));
  Val x682 = (((x621 + (x622 * Val(2))) + (x623 * Val(4))) + (x624 * Val(8)));
  Val x683 = (((x682 + (x625 * Val(16))) + (x626 * Val(32))) + (x627 * Val(64)));
  Val x684 = (((x683 + (x628 * Val(128))) + (x629 * Val(256))) + (x630 * Val(512)));
  Val x685 = (((x684 + (x631 * Val(1024))) + (x632 * Val(2048))) + (x633 * Val(4096)));
  Val x686 = (((x685 + (x634 * Val(8192))) + (x635 * Val(16384))) + (x636 * Val(32768)));
  Val x687 = (((x572 + (x573 * Val(2))) + (x574 * Val(4))) + (x575 * Val(8)));
  Val x688 = (((x687 + (x576 * Val(16))) + (x577 * Val(32))) + (x578 * Val(64)));
  Val x689 = (((x688 + (x579 * Val(128))) + (x580 * Val(256))) + (x581 * Val(512)));
  Val x690 = (((x689 + (x582 * Val(1024))) + (x583 * Val(2048))) + (x584 * Val(4096)));
  Val x691 = (((x690 + (x585 * Val(8192))) + (x586 * Val(16384))) + (x587 * Val(32768)));
  Val x692 = (((x588 + (x589 * Val(2))) + (x590 * Val(4))) + (x591 * Val(8)));
  Val x693 = (((x692 + (x592 * Val(16))) + (x593 * Val(32))) + (x594 * Val(64)));
  Val x694 = (((x693 + (x595 * Val(128))) + (x596 * Val(256))) + (x597 * Val(512)));
  Val x695 = (((x694 + (x598 * Val(1024))) + (x599 * Val(2048))) + (x600 * Val(4096)));
  Val x696 = (((x695 + (x601 * Val(8192))) + (x602 * Val(16384))) + (x603 * Val(32768)));
  Val x697 = (((x539 + (x540 * Val(2))) + (x541 * Val(4))) + (x542 * Val(8)));
  Val x698 = (((x697 + (x543 * Val(16))) + (x544 * Val(32))) + (x545 * Val(64)));
  Val x699 = (((x698 + (x546 * Val(128))) + (x547 * Val(256))) + (x548 * Val(512)));
  Val x700 = (((x699 + (x549 * Val(1024))) + (x550 * Val(2048))) + (x551 * Val(4096)));
  Val x701 = (((x700 + (x552 * Val(8192))) + (x553 * Val(16384))) + (x554 * Val(32768)));
  Val x702 = (((x555 + (x556 * Val(2))) + (x557 * Val(4))) + (x558 * Val(8)));
  Val x703 = (((x702 + (x559 * Val(16))) + (x560 * Val(32))) + (x561 * Val(64)));
  Val x704 = (((x703 + (x562 * Val(128))) + (x563 * Val(256))) + (x564 * Val(512)));
  Val x705 = (((x704 + (x565 * Val(1024))) + (x566 * Val(2048))) + (x567 * Val(4096)));
  Val x706 = (((x705 + (x568 * Val(8192))) + (x569 * Val(16384))) + (x570 * Val(32768)));
  Val x707 = (((x506 + (x507 * Val(2))) + (x508 * Val(4))) + (x509 * Val(8)));
  Val x708 = (((x707 + (x510 * Val(16))) + (x511 * Val(32))) + (x512 * Val(64)));
  Val x709 = (((x708 + (x513 * Val(128))) + (x514 * Val(256))) + (x515 * Val(512)));
  Val x710 = (((x709 + (x516 * Val(1024))) + (x517 * Val(2048))) + (x518 * Val(4096)));
  Val x711 = (((x710 + (x519 * Val(8192))) + (x520 * Val(16384))) + (x521 * Val(32768)));
  Val x712 = (((x522 + (x523 * Val(2))) + (x524 * Val(4))) + (x525 * Val(8)));
  Val x713 = (((x712 + (x526 * Val(16))) + (x527 * Val(32))) + (x528 * Val(64)));
  Val x714 = (((x713 + (x529 * Val(128))) + (x530 * Val(256))) + (x531 * Val(512)));
  Val x715 = (((x714 + (x532 * Val(1024))) + (x533 * Val(2048))) + (x534 * Val(4096)));
  Val x716 = (((x715 + (x535 * Val(8192))) + (x536 * Val(16384))) + (x537 * Val(32768)));
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:235)
  TopStateKflatStruct100Array x717 = arg0.kflat;
  Val x718 = x717[0]._super._super;
  Val x719 = x717[1]._super._super;
  Val x720 = x717[2]._super._super;
  Val x721 = x717[3]._super._super;
  Val x722 = x717[4]._super._super;
  Val x723 = x717[5]._super._super;
  Val x724 = x717[6]._super._super;
  Val x725 = x717[7]._super._super;
  Val x726 = x717[8]._super._super;
  Val x727 = x717[9]._super._super;
  Val x728 = x717[10]._super._super;
  Val x729 = x717[11]._super._super;
  Val x730 = x717[12]._super._super;
  Val x731 = x717[13]._super._super;
  Val x732 = x717[14]._super._super;
  Val x733 = x717[15]._super._super;
  Val x734 = x717[16]._super._super;
  Val x735 = x717[17]._super._super;
  Val x736 = x717[18]._super._super;
  Val x737 = x717[19]._super._super;
  Val x738 = x717[20]._super._super;
  Val x739 = x717[21]._super._super;
  Val x740 = x717[22]._super._super;
  Val x741 = x717[23]._super._super;
  Val x742 = x717[24]._super._super;
  Val x743 = x717[25]._super._super;
  Val x744 = x717[26]._super._super;
  Val x745 = x717[27]._super._super;
  Val x746 = x717[28]._super._super;
  Val x747 = x717[29]._super._super;
  Val x748 = x717[30]._super._super;
  Val x749 = x717[31]._super._super;
  Val x750 = x717[32]._super._super;
  Val x751 = x717[33]._super._super;
  Val x752 = x717[34]._super._super;
  Val x753 = x717[35]._super._super;
  Val x754 = x717[36]._super._super;
  Val x755 = x717[37]._super._super;
  Val x756 = x717[38]._super._super;
  Val x757 = x717[39]._super._super;
  Val x758 = x717[40]._super._super;
  Val x759 = x717[41]._super._super;
  Val x760 = x717[42]._super._super;
  Val x761 = x717[43]._super._super;
  Val x762 = x717[44]._super._super;
  Val x763 = x717[45]._super._super;
  Val x764 = x717[46]._super._super;
  Val x765 = x717[47]._super._super;
  Val x766 = x717[48]._super._super;
  Val x767 = x717[49]._super._super;
  Val x768 = x717[50]._super._super;
  Val x769 = x717[51]._super._super;
  Val x770 = x717[52]._super._super;
  Val x771 = x717[53]._super._super;
  Val x772 = x717[54]._super._super;
  Val x773 = x717[55]._super._super;
  Val x774 = x717[56]._super._super;
  Val x775 = x717[57]._super._super;
  Val x776 = x717[58]._super._super;
  Val x777 = x717[59]._super._super;
  Val x778 = x717[60]._super._super;
  Val x779 = x717[61]._super._super;
  Val x780 = x717[62]._super._super;
  Val x781 = x717[63]._super._super;
  Val x782 = x717[64]._super._super;
  Val x783 = x717[65]._super._super;
  Val x784 = x717[66]._super._super;
  Val x785 = x717[67]._super._super;
  Val x786 = x717[68]._super._super;
  Val x787 = x717[69]._super._super;
  Val x788 = x717[70]._super._super;
  Val x789 = x717[71]._super._super;
  Val x790 = x717[72]._super._super;
  Val x791 = x717[73]._super._super;
  Val x792 = x717[74]._super._super;
  Val x793 = x717[75]._super._super;
  Val x794 = x717[76]._super._super;
  Val x795 = x717[77]._super._super;
  Val x796 = x717[78]._super._super;
  Val x797 = x717[79]._super._super;
  Val x798 = x717[80]._super._super;
  Val x799 = x717[81]._super._super;
  Val x800 = x717[82]._super._super;
  Val x801 = x717[83]._super._super;
  Val x802 = x717[84]._super._super;
  Val x803 = x717[85]._super._super;
  Val x804 = x717[86]._super._super;
  Val x805 = x717[87]._super._super;
  Val x806 = x717[88]._super._super;
  Val x807 = x717[89]._super._super;
  Val x808 = x717[90]._super._super;
  Val x809 = x717[91]._super._super;
  Val x810 = x717[92]._super._super;
  Val x811 = x717[93]._super._super;
  Val x812 = x717[94]._super._super;
  Val x813 = x717[95]._super._super;
  Val x814 = x717[96]._super._super;
  Val x815 = x717[97]._super._super;
  Val x816 = x717[98]._super._super;
  Val x817 = x717[99]._super._super;
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:262)
  TopStateStruct x818 = exec_TopState(
      ctx,
      Val800Array{
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), x374,   x375,   x376,   x377,
          x378,   x379,   x380,   x381,   x382,   x383,   x384,   x385,   x386,   x387,   x388,
          x389,   x390,   x391,   x392,   x393,   x394,   x395,   x396,   x397,   x398,   x399,
          x400,   x401,   x402,   x403,   x404,   x405,   x407,   x408,   x409,   x410,   x411,
          x412,   x413,   x414,   x415,   x416,   x417,   x418,   x419,   x420,   x421,   x422,
          x423,   x424,   x425,   x426,   x427,   x428,   x429,   x430,   x431,   x432,   x433,
          x434,   x435,   x436,   x437,   x438,   x440,   x441,   x442,   x443,   x444,   x445,
          x446,   x447,   x448,   x449,   x450,   x451,   x452,   x453,   x454,   x455,   x456,
          x457,   x458,   x459,   x460,   x461,   x462,   x463,   x464,   x465,   x466,   x467,
          x468,   x469,   x470,   x471,   x473,   x474,   x475,   x476,   x477,   x478,   x479,
          x480,   x481,   x482,   x483,   x484,   x485,   x486,   x487,   x488,   x489,   x490,
          x491,   x492,   x493,   x494,   x495,   x496,   x497,   x498,   x499,   x500,   x501,
          x502,   x503,   x504,   Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), x506,
          x507,   x508,   x509,   x510,   x511,   x512,   x513,   x514,   x515,   x516,   x517,
          x518,   x519,   x520,   x521,   x522,   x523,   x524,   x525,   x526,   x527,   x528,
          x529,   x530,   x531,   x532,   x533,   x534,   x535,   x536,   x537,   x539,   x540,
          x541,   x542,   x543,   x544,   x545,   x546,   x547,   x548,   x549,   x550,   x551,
          x552,   x553,   x554,   x555,   x556,   x557,   x558,   x559,   x560,   x561,   x562,
          x563,   x564,   x565,   x566,   x567,   x568,   x569,   x570,   x572,   x573,   x574,
          x575,   x576,   x577,   x578,   x579,   x580,   x581,   x582,   x583,   x584,   x585,
          x586,   x587,   x588,   x589,   x590,   x591,   x592,   x593,   x594,   x595,   x596,
          x597,   x598,   x599,   x600,   x601,   x602,   x603,   x605,   x606,   x607,   x608,
          x609,   x610,   x611,   x612,   x613,   x614,   x615,   x616,   x617,   x618,   x619,
          x620,   x621,   x622,   x623,   x624,   x625,   x626,   x627,   x628,   x629,   x630,
          x631,   x632,   x633,   x634,   x635,   x636,   Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val100Array{x718, x719, x720, x721, x722, x723, x724, x725, x726, x727, x728, x729, x730,
                  x731, x732, x733, x734, x735, x736, x737, x738, x739, x740, x741, x742, x743,
                  x744, x745, x746, x747, x748, x749, x750, x751, x752, x753, x754, x755, x756,
                  x757, x758, x759, x760, x761, x762, x763, x764, x765, x766, x767, x768, x769,
                  x770, x771, x772, x773, x774, x775, x776, x777, x778, x779, x780, x781, x782,
                  x783, x784, x785, x786, x787, x788, x789, x790, x791, x792, x793, x794, x795,
                  x796, x797, x798, x799, x800, x801, x802, x803, x804, x805, x806, x807, x808,
                  x809, x810, x811, x812, x813, x814, x815, x816, x817},
      Val16Array{x641,
                 x646,
                 x651,
                 x656,
                 x661,
                 x666,
                 x671,
                 x676,
                 x681,
                 x686,
                 x691,
                 x696,
                 x701,
                 x706,
                 x711,
                 x716},
      LAYOUT_LOOKUP(layout1, _super));
  return x818;
}

} // namespace risc0::impl
