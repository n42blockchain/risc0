#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

<<<<<<< HEAD
__device__ void step_Top_8(ExecContext& ctx, MutableBuf arg0) {
  // ControlState(zirgen/circuit/keccak2/top.zir:403)
  // ComputeCurrentStep(zirgen/circuit/keccak2/top.zir:458)
  set(ctx, arg0, 12, Val(6));
  // ControlState(zirgen/circuit/keccak2/top.zir:404)
  set(ctx, arg0, 13, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:405)
  set(ctx, arg0, 14, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:406)
  set(ctx, arg0, 15, get(ctx, arg0, 15, 1));
  return;
}
__device__ void step_Top_24(ExecContext& ctx, MutableBuf arg0) {
  if (to_size_t(get(ctx, arg0, 0, 1))) {
    // ComputeCurrentStep(zirgen/circuit/keccak2/top.zir:448)
    step_Top_1(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 1, 1))) {
    step_Top_2(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 2, 1))) {
    step_Top_5(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 3, 1))) {
    step_Top_6(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 4, 1))) {
    step_Top_7(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 5, 1))) {
    step_Top_8(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 6, 1))) {
    step_Top_9(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 7, 1))) {
    step_Top_10(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 8, 1))) {
    step_Top_13(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 9, 1))) {
    step_Top_16(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 10, 1))) {
    step_Top_22(ctx, arg0);
  }
  if (to_size_t(get(ctx, arg0, 11, 1))) {
    step_Top_23(ctx, arg0);
  }
  return;
=======
__device__ OneHot_24_Struct exec_OneHot_24_(ExecContext& ctx,
                                            Val arg0,
                                            BoundLayout<OneHot_24_Layout> layout1) {
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:7)
  NondetBitRegStruct24Array x2 = map(
      Val24Array{Val(0),  Val(1),  Val(2),  Val(3),  Val(4),  Val(5),  Val(6),  Val(7),
                 Val(8),  Val(9),  Val(10), Val(11), Val(12), Val(13), Val(14), Val(15),
                 Val(16), Val(17), Val(18), Val(19), Val(20), Val(21), Val(22), Val(23)},
      LAYOUT_LOOKUP(layout1, _super),
      ([&](Val24Array::value_type x3, BoundLayout<NondetRegLayout24LayoutArray::value_type> x4) {
        NondetBitRegStruct x5 = exec_NondetBitReg(ctx, isz((x3 - arg0)), x4);
        return x5;
      }));
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:9)
  Val x6 = (x2[0]._super._super + x2[1]._super._super);
  Val x7 = ((x6 + x2[2]._super._super) + x2[3]._super._super);
  Val x8 = ((x7 + x2[4]._super._super) + x2[5]._super._super);
  Val x9 = ((x8 + x2[6]._super._super) + x2[7]._super._super);
  Val x10 = ((x9 + x2[8]._super._super) + x2[9]._super._super);
  Val x11 = ((x10 + x2[10]._super._super) + x2[11]._super._super);
  Val x12 = ((x11 + x2[12]._super._super) + x2[13]._super._super);
  Val x13 = ((x12 + x2[14]._super._super) + x2[15]._super._super);
  Val x14 = ((x13 + x2[16]._super._super) + x2[17]._super._super);
  Val x15 = ((x14 + x2[18]._super._super) + x2[19]._super._super);
  Val x16 = ((x15 + x2[20]._super._super) + x2[21]._super._super);
  Val x17 = ((x16 + x2[22]._super._super) + x2[23]._super._super);
  EQZ((x17 - Val(1)), "OneHot(zirgen/circuit/keccak2/one_hot.zir:9)");
  // OneHot(zirgen/circuit/keccak2/one_hot.zir:11)
  Val x18 = (x2[2]._super._super * Val(2));
  Val x19 = (x2[3]._super._super * Val(3));
  Val x20 = (x2[4]._super._super * Val(4));
  Val x21 = (x2[5]._super._super * Val(5));
  Val x22 = (x2[6]._super._super * Val(6));
  Val x23 = (x2[7]._super._super * Val(7));
  Val x24 = (x2[8]._super._super * Val(8));
  Val x25 = (x2[9]._super._super * Val(9));
  Val x26 = (x2[10]._super._super * Val(10));
  Val x27 = (x2[11]._super._super * Val(11));
  Val x28 = (x2[12]._super._super * Val(12));
  Val x29 = (x2[13]._super._super * Val(13));
  Val x30 = (x2[14]._super._super * Val(14));
  Val x31 = (x2[15]._super._super * Val(15));
  Val x32 = (x2[16]._super._super * Val(16));
  Val x33 = (x2[17]._super._super * Val(17));
  Val x34 = (x2[18]._super._super * Val(18));
  Val x35 = (x2[19]._super._super * Val(19));
  Val x36 = (x2[20]._super._super * Val(20));
  Val x37 = (x2[21]._super._super * Val(21));
  Val x38 = (x2[22]._super._super * Val(22));
  Val x39 = (x2[23]._super._super * Val(23));
  Val x40 = (x2[1]._super._super + x18);
  Val x41 = (((x40 + x19) + x20) + x21);
  Val x42 = (((x41 + x22) + x23) + x24);
  Val x43 = (((x42 + x25) + x26) + x27);
  Val x44 = (((x43 + x28) + x29) + x30);
  Val x45 = (((x44 + x31) + x32) + x33);
  Val x46 = (((x45 + x34) + x35) + x36);
  Val x47 = (((x46 + x37) + x38) + x39);
  EQZ((x47 - arg0), "OneHot(zirgen/circuit/keccak2/one_hot.zir:11)");
  return OneHot_24_Struct{._super = x2};
}
__device__ TopStateStruct exec_InitCycle(ExecContext& ctx, BoundLayout<TopStateLayout> layout0) {
  // Log(<preamble>:22)
  // InitCycle(zirgen/circuit/keccak2/top.zir:305)
  INVOKE_EXTERN(ctx, log, "InitCycle", std::initializer_list<Val>{});
  // InitCycle(zirgen/circuit/keccak2/top.zir:314)
  TopStateStruct x1 = exec_TopState(
      ctx,
      Val800Array{
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val100Array{Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val16Array{Val(58983),
                 Val(27145),
                 Val(44677),
                 Val(47975),
                 Val(62322),
                 Val(15470),
                 Val(62778),
                 Val(42319),
                 Val(21119),
                 Val(20750),
                 Val(26764),
                 Val(39685),
                 Val(55723),
                 Val(8067),
                 Val(52505),
                 Val(23520)},
      layout0);
  return x1;
}
__device__ WrapOneHotStruct exec_WrapOneHot(ExecContext& ctx,
                                            OneHot_12_Struct arg0,
                                            BoundLayout<WrapOneHotLayout> layout1) {
  // WrapOneHot(zirgen/circuit/keccak2/top.zir:468)
  WrapOneHot_SuperStruct12Array x2 = map(
      Val12Array{Val(0),
                 Val(1),
                 Val(2),
                 Val(3),
                 Val(4),
                 Val(5),
                 Val(6),
                 Val(7),
                 Val(8),
                 Val(9),
                 Val(10),
                 Val(11)},
      LAYOUT_LOOKUP(layout1, _super),
      ([&](Val12Array::value_type x3, BoundLayout<NondetRegLayout12LayoutArray::value_type> x4) {
        NondetRegStruct x5 = exec_Reg(ctx, arg0.bits[to_size_t(x3)]._super._super._super, x4);
        return WrapOneHot_SuperStruct{._super = x5};
      }));
  return WrapOneHotStruct{._super = x2};
>>>>>>> origin/main
}

} // namespace risc0::circuit::keccak::cuda
