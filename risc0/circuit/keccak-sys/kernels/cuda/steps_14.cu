#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ TopStateStruct back_TopState(ExecContext& ctx,
                                        Index distance0,
                                        BoundLayout<TopStateLayout> layout1) {
  // TopState(zirgen/circuit/keccak2/top.zir:35)
  TopStateBitsStruct800Array x2 = map(
      Val800Array{
          Val(0),   Val(1),   Val(2),   Val(3),   Val(4),   Val(5),   Val(6),   Val(7),   Val(8),
          Val(9),   Val(10),  Val(11),  Val(12),  Val(13),  Val(14),  Val(15),  Val(16),  Val(17),
          Val(18),  Val(19),  Val(20),  Val(21),  Val(22),  Val(23),  Val(24),  Val(25),  Val(26),
          Val(27),  Val(28),  Val(29),  Val(30),  Val(31),  Val(32),  Val(33),  Val(34),  Val(35),
          Val(36),  Val(37),  Val(38),  Val(39),  Val(40),  Val(41),  Val(42),  Val(43),  Val(44),
          Val(45),  Val(46),  Val(47),  Val(48),  Val(49),  Val(50),  Val(51),  Val(52),  Val(53),
          Val(54),  Val(55),  Val(56),  Val(57),  Val(58),  Val(59),  Val(60),  Val(61),  Val(62),
          Val(63),  Val(64),  Val(65),  Val(66),  Val(67),  Val(68),  Val(69),  Val(70),  Val(71),
          Val(72),  Val(73),  Val(74),  Val(75),  Val(76),  Val(77),  Val(78),  Val(79),  Val(80),
          Val(81),  Val(82),  Val(83),  Val(84),  Val(85),  Val(86),  Val(87),  Val(88),  Val(89),
          Val(90),  Val(91),  Val(92),  Val(93),  Val(94),  Val(95),  Val(96),  Val(97),  Val(98),
          Val(99),  Val(100), Val(101), Val(102), Val(103), Val(104), Val(105), Val(106), Val(107),
          Val(108), Val(109), Val(110), Val(111), Val(112), Val(113), Val(114), Val(115), Val(116),
          Val(117), Val(118), Val(119), Val(120), Val(121), Val(122), Val(123), Val(124), Val(125),
          Val(126), Val(127), Val(128), Val(129), Val(130), Val(131), Val(132), Val(133), Val(134),
          Val(135), Val(136), Val(137), Val(138), Val(139), Val(140), Val(141), Val(142), Val(143),
          Val(144), Val(145), Val(146), Val(147), Val(148), Val(149), Val(150), Val(151), Val(152),
          Val(153), Val(154), Val(155), Val(156), Val(157), Val(158), Val(159), Val(160), Val(161),
          Val(162), Val(163), Val(164), Val(165), Val(166), Val(167), Val(168), Val(169), Val(170),
          Val(171), Val(172), Val(173), Val(174), Val(175), Val(176), Val(177), Val(178), Val(179),
          Val(180), Val(181), Val(182), Val(183), Val(184), Val(185), Val(186), Val(187), Val(188),
          Val(189), Val(190), Val(191), Val(192), Val(193), Val(194), Val(195), Val(196), Val(197),
          Val(198), Val(199), Val(200), Val(201), Val(202), Val(203), Val(204), Val(205), Val(206),
          Val(207), Val(208), Val(209), Val(210), Val(211), Val(212), Val(213), Val(214), Val(215),
          Val(216), Val(217), Val(218), Val(219), Val(220), Val(221), Val(222), Val(223), Val(224),
          Val(225), Val(226), Val(227), Val(228), Val(229), Val(230), Val(231), Val(232), Val(233),
          Val(234), Val(235), Val(236), Val(237), Val(238), Val(239), Val(240), Val(241), Val(242),
          Val(243), Val(244), Val(245), Val(246), Val(247), Val(248), Val(249), Val(250), Val(251),
          Val(252), Val(253), Val(254), Val(255), Val(256), Val(257), Val(258), Val(259), Val(260),
          Val(261), Val(262), Val(263), Val(264), Val(265), Val(266), Val(267), Val(268), Val(269),
          Val(270), Val(271), Val(272), Val(273), Val(274), Val(275), Val(276), Val(277), Val(278),
          Val(279), Val(280), Val(281), Val(282), Val(283), Val(284), Val(285), Val(286), Val(287),
          Val(288), Val(289), Val(290), Val(291), Val(292), Val(293), Val(294), Val(295), Val(296),
          Val(297), Val(298), Val(299), Val(300), Val(301), Val(302), Val(303), Val(304), Val(305),
          Val(306), Val(307), Val(308), Val(309), Val(310), Val(311), Val(312), Val(313), Val(314),
          Val(315), Val(316), Val(317), Val(318), Val(319), Val(320), Val(321), Val(322), Val(323),
          Val(324), Val(325), Val(326), Val(327), Val(328), Val(329), Val(330), Val(331), Val(332),
          Val(333), Val(334), Val(335), Val(336), Val(337), Val(338), Val(339), Val(340), Val(341),
          Val(342), Val(343), Val(344), Val(345), Val(346), Val(347), Val(348), Val(349), Val(350),
          Val(351), Val(352), Val(353), Val(354), Val(355), Val(356), Val(357), Val(358), Val(359),
          Val(360), Val(361), Val(362), Val(363), Val(364), Val(365), Val(366), Val(367), Val(368),
          Val(369), Val(370), Val(371), Val(372), Val(373), Val(374), Val(375), Val(376), Val(377),
          Val(378), Val(379), Val(380), Val(381), Val(382), Val(383), Val(384), Val(385), Val(386),
          Val(387), Val(388), Val(389), Val(390), Val(391), Val(392), Val(393), Val(394), Val(395),
          Val(396), Val(397), Val(398), Val(399), Val(400), Val(401), Val(402), Val(403), Val(404),
          Val(405), Val(406), Val(407), Val(408), Val(409), Val(410), Val(411), Val(412), Val(413),
          Val(414), Val(415), Val(416), Val(417), Val(418), Val(419), Val(420), Val(421), Val(422),
          Val(423), Val(424), Val(425), Val(426), Val(427), Val(428), Val(429), Val(430), Val(431),
          Val(432), Val(433), Val(434), Val(435), Val(436), Val(437), Val(438), Val(439), Val(440),
          Val(441), Val(442), Val(443), Val(444), Val(445), Val(446), Val(447), Val(448), Val(449),
          Val(450), Val(451), Val(452), Val(453), Val(454), Val(455), Val(456), Val(457), Val(458),
          Val(459), Val(460), Val(461), Val(462), Val(463), Val(464), Val(465), Val(466), Val(467),
          Val(468), Val(469), Val(470), Val(471), Val(472), Val(473), Val(474), Val(475), Val(476),
          Val(477), Val(478), Val(479), Val(480), Val(481), Val(482), Val(483), Val(484), Val(485),
          Val(486), Val(487), Val(488), Val(489), Val(490), Val(491), Val(492), Val(493), Val(494),
          Val(495), Val(496), Val(497), Val(498), Val(499), Val(500), Val(501), Val(502), Val(503),
          Val(504), Val(505), Val(506), Val(507), Val(508), Val(509), Val(510), Val(511), Val(512),
          Val(513), Val(514), Val(515), Val(516), Val(517), Val(518), Val(519), Val(520), Val(521),
          Val(522), Val(523), Val(524), Val(525), Val(526), Val(527), Val(528), Val(529), Val(530),
          Val(531), Val(532), Val(533), Val(534), Val(535), Val(536), Val(537), Val(538), Val(539),
          Val(540), Val(541), Val(542), Val(543), Val(544), Val(545), Val(546), Val(547), Val(548),
          Val(549), Val(550), Val(551), Val(552), Val(553), Val(554), Val(555), Val(556), Val(557),
          Val(558), Val(559), Val(560), Val(561), Val(562), Val(563), Val(564), Val(565), Val(566),
          Val(567), Val(568), Val(569), Val(570), Val(571), Val(572), Val(573), Val(574), Val(575),
          Val(576), Val(577), Val(578), Val(579), Val(580), Val(581), Val(582), Val(583), Val(584),
          Val(585), Val(586), Val(587), Val(588), Val(589), Val(590), Val(591), Val(592), Val(593),
          Val(594), Val(595), Val(596), Val(597), Val(598), Val(599), Val(600), Val(601), Val(602),
          Val(603), Val(604), Val(605), Val(606), Val(607), Val(608), Val(609), Val(610), Val(611),
          Val(612), Val(613), Val(614), Val(615), Val(616), Val(617), Val(618), Val(619), Val(620),
          Val(621), Val(622), Val(623), Val(624), Val(625), Val(626), Val(627), Val(628), Val(629),
          Val(630), Val(631), Val(632), Val(633), Val(634), Val(635), Val(636), Val(637), Val(638),
          Val(639), Val(640), Val(641), Val(642), Val(643), Val(644), Val(645), Val(646), Val(647),
          Val(648), Val(649), Val(650), Val(651), Val(652), Val(653), Val(654), Val(655), Val(656),
          Val(657), Val(658), Val(659), Val(660), Val(661), Val(662), Val(663), Val(664), Val(665),
          Val(666), Val(667), Val(668), Val(669), Val(670), Val(671), Val(672), Val(673), Val(674),
          Val(675), Val(676), Val(677), Val(678), Val(679), Val(680), Val(681), Val(682), Val(683),
          Val(684), Val(685), Val(686), Val(687), Val(688), Val(689), Val(690), Val(691), Val(692),
          Val(693), Val(694), Val(695), Val(696), Val(697), Val(698), Val(699), Val(700), Val(701),
          Val(702), Val(703), Val(704), Val(705), Val(706), Val(707), Val(708), Val(709), Val(710),
          Val(711), Val(712), Val(713), Val(714), Val(715), Val(716), Val(717), Val(718), Val(719),
          Val(720), Val(721), Val(722), Val(723), Val(724), Val(725), Val(726), Val(727), Val(728),
          Val(729), Val(730), Val(731), Val(732), Val(733), Val(734), Val(735), Val(736), Val(737),
          Val(738), Val(739), Val(740), Val(741), Val(742), Val(743), Val(744), Val(745), Val(746),
          Val(747), Val(748), Val(749), Val(750), Val(751), Val(752), Val(753), Val(754), Val(755),
          Val(756), Val(757), Val(758), Val(759), Val(760), Val(761), Val(762), Val(763), Val(764),
          Val(765), Val(766), Val(767), Val(768), Val(769), Val(770), Val(771), Val(772), Val(773),
          Val(774), Val(775), Val(776), Val(777), Val(778), Val(779), Val(780), Val(781), Val(782),
          Val(783), Val(784), Val(785), Val(786), Val(787), Val(788), Val(789), Val(790), Val(791),
          Val(792), Val(793), Val(794), Val(795), Val(796), Val(797), Val(798), Val(799)},
      LAYOUT_LOOKUP(layout1, bits),
      ([&](Val800Array::value_type x3, BoundLayout<NondetRegLayout800LayoutArray::value_type> x4) {
        // TopState(zirgen/circuit/keccak2/top.zir:36)
        NondetRegStruct x5 = back_Reg(ctx, distance0, x4);
        return TopStateBitsStruct{._super = x5};
      }));
  // TopState(zirgen/circuit/keccak2/top.zir:39)
  TopStateKflatStruct100Array x6 = map(
      Val100Array{
          Val(0),  Val(1),  Val(2),  Val(3),  Val(4),  Val(5),  Val(6),  Val(7),  Val(8),  Val(9),
          Val(10), Val(11), Val(12), Val(13), Val(14), Val(15), Val(16), Val(17), Val(18), Val(19),
          Val(20), Val(21), Val(22), Val(23), Val(24), Val(25), Val(26), Val(27), Val(28), Val(29),
          Val(30), Val(31), Val(32), Val(33), Val(34), Val(35), Val(36), Val(37), Val(38), Val(39),
          Val(40), Val(41), Val(42), Val(43), Val(44), Val(45), Val(46), Val(47), Val(48), Val(49),
          Val(50), Val(51), Val(52), Val(53), Val(54), Val(55), Val(56), Val(57), Val(58), Val(59),
          Val(60), Val(61), Val(62), Val(63), Val(64), Val(65), Val(66), Val(67), Val(68), Val(69),
          Val(70), Val(71), Val(72), Val(73), Val(74), Val(75), Val(76), Val(77), Val(78), Val(79),
          Val(80), Val(81), Val(82), Val(83), Val(84), Val(85), Val(86), Val(87), Val(88), Val(89),
          Val(90), Val(91), Val(92), Val(93), Val(94), Val(95), Val(96), Val(97), Val(98), Val(99)},
      LAYOUT_LOOKUP(layout1, kflat),
      ([&](Val100Array::value_type x7, BoundLayout<NondetRegLayout100LayoutArray::value_type> x8) {
        // TopState(zirgen/circuit/keccak2/top.zir:40)
        NondetRegStruct x9 = back_Reg(ctx, distance0, x8);
        return TopStateKflatStruct{._super = x9};
      }));
  // TopState(zirgen/circuit/keccak2/top.zir:42)
  NondetRegStruct16Array x10 = map(
      Val16Array{Val(0),
                 Val(1),
                 Val(2),
                 Val(3),
                 Val(4),
                 Val(5),
                 Val(6),
                 Val(7),
                 Val(8),
                 Val(9),
                 Val(10),
                 Val(11),
                 Val(12),
                 Val(13),
                 Val(14),
                 Val(15)},
      LAYOUT_LOOKUP(layout1, sflat),
      ([&](Val16Array::value_type x11, BoundLayout<NondetRegLayout16LayoutArray::value_type> x12) {
        // TopState(zirgen/circuit/keccak2/top.zir:43)
        NondetRegStruct x13 = back_Reg(ctx, distance0, x12);
        return x13;
      }));
  return TopStateStruct{.bits = x2, .kflat = x6, .sflat = x10};
}
__device__ TopStateStruct exec_ShutdownCycle(ExecContext& ctx,
                                             TopStateStruct arg0,
                                             BoundLayout<TopStateLayout> layout1,
                                             GlobalBuf global2) {
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:394)
  BoundLayout<_globalLayout> x3 = BIND_LAYOUT(kLayoutGlobal, global2);
  // Log(<preamble>:22)
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:395)
  INVOKE_EXTERN(ctx, log, "ShutdownCycle", std::initializer_list<Val>{});
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:394)
  Val16Array x4 = Val16Array{arg0.sflat[0]._super,
                             arg0.sflat[1]._super,
                             arg0.sflat[2]._super,
                             arg0.sflat[3]._super,
                             arg0.sflat[4]._super,
                             arg0.sflat[5]._super,
                             arg0.sflat[6]._super,
                             arg0.sflat[7]._super,
                             arg0.sflat[8]._super,
                             arg0.sflat[9]._super,
                             arg0.sflat[10]._super,
                             arg0.sflat[11]._super,
                             arg0.sflat[12]._super,
                             arg0.sflat[13]._super,
                             arg0.sflat[14]._super,
                             arg0.sflat[15]._super};
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:396)
  DigestRegStruct x5 = exec_DigestReg(ctx, x4, LAYOUT_LOOKUP(x3, finalDigest));
  // ShutdownCycle(zirgen/circuit/keccak2/top.zir:399)
  TopStateStruct x6 = exec_TopState(
      ctx,
      Val800Array{
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
          Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      Val100Array{Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0),
                  Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0), Val(0)},
      x4,
      layout1);
  return x6;
}

} // namespace risc0::circuit::keccak::cuda
