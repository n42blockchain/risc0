#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"

namespace risc0::impl {

__device__ TopStruct exec_Top(ExecContext& ctx, BoundLayout<TopLayout> layout0, GlobalBuf global1) {
  // Top(zirgen/circuit/keccak2/top.zir:477)
  BoundLayout<WrapOneHotLayout> x2 = LAYOUT_LOOKUP(layout0, cycleMux);
  // Top(zirgen/circuit/keccak2/top.zir:478)
  BoundLayout<ControlStateLayout> x3 = LAYOUT_LOOKUP(layout0, controlState);
  // Top(zirgen/circuit/keccak2/top.zir:479)
  BoundLayout<TopControlStateLayout> x4 = LAYOUT_LOOKUP(layout0, controlStateRedef);
  // Top(zirgen/circuit/keccak2/top.zir:490)
  BoundLayout<TopStateLayout> x5 = LAYOUT_LOOKUP(layout0, curState);
  // Top(zirgen/circuit/keccak2/top.zir:491)
  BoundLayout<TopCurStateLayout> x6 = LAYOUT_LOOKUP(layout0, curStateRedef);
  // IsFirstCycle(zirgen/circuit/keccak2/top.zir:278)
  // Top(zirgen/circuit/keccak2/top.zir:476)
  Val x7 = INVOKE_EXTERN(ctx, isFirstCycle);
  NondetRegStruct x8 = exec_NondetReg(ctx, x7, LAYOUT_LOOKUP(layout0, isFirst));
  Val x9 = x8._super;
  // Top(zirgen/circuit/keccak2/top.zir:479)
  BoundLayout<ControlStateLayout> x10 = LAYOUT_LOOKUP(x4, _super);
  ControlStateStruct x11;
  if (to_size_t(x9)) {
    // Top(zirgen/circuit/keccak2/top.zir:480)
    ControlStateStruct x12 = back_ControlState(ctx, 1, x3);
    EQZ(x12.cycleType._super, "Top(zirgen/circuit/keccak2/top.zir:480)");
    // Top(zirgen/circuit/keccak2/top.zir:481)
    ControlStateStruct x13 =
        exec_ControlState(ctx, Val(11), Val(0), Val(0), Val(0), LAYOUT_LOOKUP(x4, arm0));
    x11 = x13;
  } else if (to_size_t((Val(1) - x9))) {
    // Top(zirgen/circuit/keccak2/top.zir:483)
    WrapOneHotStruct x14 = back_WrapOneHot(ctx, 1, x2);
    ControlStateStruct x15 = back_ControlState(ctx, 1, x3);
    WrapOneHot_SuperStruct12Array x16 = x14._super;
    Val x17 = x16[0]._super._super;
    Val x18 = x16[1]._super._super;
    Val x19 = x16[2]._super._super;
    Val x20 = x16[3]._super._super;
    Val x21 = x16[4]._super._super;
    Val x22 = x16[5]._super._super;
    Val x23 = x16[6]._super._super;
    Val x24 = x16[7]._super._super;
    Val x25 = x16[8]._super._super;
    Val x26 = x16[9]._super._super;
    Val x27 = x16[10]._super._super;
    Val x28 = x16[11]._super._super;
    ControlStateStruct x29 = exec_ComputeCurrentStep(
        ctx,
        Val12Array{x17, x18, x19, x20, x21, x22, x23, x24, x25, x26, x27, x28},
        x15,
        LAYOUT_LOOKUP(x4, arm1));
    x11 = x29;
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  // Top(zirgen/circuit/keccak2/top.zir:479)
  ControlStateStruct x30 = back_ControlState(ctx, 0, x10);
  // Top(zirgen/circuit/keccak2/top.zir:485)
  OneHot_12_Struct x31 = exec_OneHot_12_(ctx, x30.cycleType._super, LAYOUT_LOOKUP(layout0, oneHot));
  // Top(zirgen/circuit/keccak2/top.zir:486)
  WrapOneHotStruct x32 = exec_WrapOneHot(ctx, x31, x2);
  WrapOneHot_SuperStruct12Array x33 = x32._super;
  Val x34 = x33[0]._super._super;
  Val x35 = x33[1]._super._super;
  Val x36 = x33[2]._super._super;
  Val x37 = x33[3]._super._super;
  Val x38 = x33[4]._super._super;
  Val x39 = x33[5]._super._super;
  Val x40 = x33[6]._super._super;
  Val x41 = x33[7]._super._super;
  Val x42 = x33[8]._super._super;
  Val x43 = x33[9]._super._super;
  Val x44 = x33[10]._super._super;
  Val x45 = x33[11]._super._super;
  TopStateStruct x46;
  if (to_size_t(x34)) {
    // Top(zirgen/circuit/keccak2/top.zir:492)
    TopStateStruct x47 = back_TopState(ctx, 1, x5);
    TopStateStruct x48 = exec_ShutdownCycle(ctx, x47, LAYOUT_LOOKUP(x6, arm0), global1);
    x46 = x48;
  } else if (to_size_t(x35)) {
    // Top(zirgen/circuit/keccak2/top.zir:493)
    TopStateStruct x49 = back_TopState(ctx, 1, x5);
    TopStateStruct x50 = exec_ReadCycle(ctx, x49, LAYOUT_LOOKUP(x6, arm1));
    x46 = x50;
  } else if (to_size_t(x36)) {
    // Top(zirgen/circuit/keccak2/top.zir:494)
    TopStateStruct x51 = back_TopState(ctx, 1, x5);
    // Top(zirgen/circuit/keccak2/top.zir:479)
    ControlStateStruct x52 = back_ControlState(ctx, 0, x10);
    // Top(zirgen/circuit/keccak2/top.zir:494)
    TopStateStruct x53 = exec_ExpandCycle(ctx, x51, x52.subType._super, LAYOUT_LOOKUP(x6, arm2));
    x46 = x53;
  } else if (to_size_t(x37)) {
    // Top(zirgen/circuit/keccak2/top.zir:495)
    TopStateStruct x54 = back_TopState(ctx, 1, x5);
    TopStateStruct x55 = back_TopState(ctx, 2, x5);
    TopStateStruct x56 = exec_WriteCycle(ctx, x54, x55, LAYOUT_LOOKUP(x6, arm3));
    x46 = x56;
  } else if (to_size_t(x38)) {
    // Top(zirgen/circuit/keccak2/top.zir:496)
    TopStateStruct x57 = back_TopState(ctx, 2, x5);
    TopStateStruct x58 = back_TopState(ctx, 1, x5);
    TopStateStruct x59 = exec_KeccakRound0(ctx, x57, x58, LAYOUT_LOOKUP(x6, arm4));
    x46 = x59;
  } else if (to_size_t(x39)) {
    // Top(zirgen/circuit/keccak2/top.zir:497)
    TopStateStruct x60 = back_TopState(ctx, 1, x5);
    TopStateStruct x61 = back_TopState(ctx, 3, x5);
    TopStateStruct x62 = back_TopState(ctx, 2, x5);
    TopStateStruct x63 = exec_KeccakRound12_0_(ctx, x60, x61, x62, x60, LAYOUT_LOOKUP(x6, arm5));
    x46 = x63;
  } else if (to_size_t(x40)) {
    // Top(zirgen/circuit/keccak2/top.zir:498)
    TopStateStruct x64 = back_TopState(ctx, 2, x5);
    TopStateStruct x65 = back_TopState(ctx, 4, x5);
    TopStateStruct x66 = back_TopState(ctx, 3, x5);
    TopStateStruct x67 = back_TopState(ctx, 1, x5);
    TopStateStruct x68 = exec_KeccakRound12_1_(ctx, x64, x65, x66, x67, LAYOUT_LOOKUP(x6, arm6));
    x46 = x68;
  } else if (to_size_t(x41)) {
    // Top(zirgen/circuit/keccak2/top.zir:499)
    TopStateStruct x69 = back_TopState(ctx, 2, x5);
    TopStateStruct x70 = back_TopState(ctx, 1, x5);
    // Top(zirgen/circuit/keccak2/top.zir:479)
    ControlStateStruct x71 = back_ControlState(ctx, 0, x10);
    // Top(zirgen/circuit/keccak2/top.zir:499)
    TopStateStruct x72 =
        exec_KeccakRound34_0_(ctx, x69, x70, x70, x71.round._super, LAYOUT_LOOKUP(x6, arm7));
    x46 = x72;
  } else if (to_size_t(x42)) {
    // Top(zirgen/circuit/keccak2/top.zir:500)
    TopStateStruct x73 = back_TopState(ctx, 3, x5);
    TopStateStruct x74 = back_TopState(ctx, 2, x5);
    TopStateStruct x75 = back_TopState(ctx, 1, x5);
    // Top(zirgen/circuit/keccak2/top.zir:479)
    ControlStateStruct x76 = back_ControlState(ctx, 0, x10);
    // Top(zirgen/circuit/keccak2/top.zir:500)
    TopStateStruct x77 =
        exec_KeccakRound34_1_(ctx, x73, x74, x75, x76.round._super, LAYOUT_LOOKUP(x6, arm8));
    x46 = x77;
  } else if (to_size_t(x43)) {
    // Top(zirgen/circuit/keccak2/top.zir:501)
    TopStateStruct x78 = back_TopState(ctx, 1, x5);
    TopStateStruct x79 = back_TopState(ctx, 2, x5);
    // Top(zirgen/circuit/keccak2/top.zir:479)
    ControlStateStruct x80 = back_ControlState(ctx, 0, x10);
    // Top(zirgen/circuit/keccak2/top.zir:501)
    TopStateStruct x81 =
        exec_ShaCycle(ctx, x78, x79, x80.round._super, x80.block._super, LAYOUT_LOOKUP(x6, arm9));
    x46 = x81;
  } else if (to_size_t(x44)) {
    // Top(zirgen/circuit/keccak2/top.zir:502)
    TopStateStruct x82 = back_TopState(ctx, 1, x5);
    TopStateStruct x83 = exec_ShaNextBlockCycle(ctx, x82, LAYOUT_LOOKUP(x6, arm10));
    x46 = x83;
  } else if (to_size_t(x45)) {
    // Top(zirgen/circuit/keccak2/top.zir:503)
    TopStateStruct x84 = exec_InitCycle(ctx, LAYOUT_LOOKUP(x6, arm11));
    x46 = x84;
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  return TopStruct{};
}

} // namespace risc0::impl
