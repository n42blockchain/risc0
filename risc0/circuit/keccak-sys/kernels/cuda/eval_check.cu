#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "eval_check.cuh"

#include "hip/hip_runtime.h"
#include "supra/fp.h"
#include <exception>
#include <iostream>

namespace risc0::circuit::keccak::cuda {

using MutableBuf = const Fp*;
using GlobalBuf = const Fp*;
using GlobalExtBuf = const FpExt*;
using ExtVal = FpExt;
using Val = Fp;
using Index = size_t;
using MixState = FpExt;

constexpr bool kDebug = false;
__constant__ FpExt poly_mix[kNumPolyMixPows];

constexpr size_t kInvRate = 4;

struct EncodedProg {
  const uint32_t* __restrict__ curPos = nullptr;

  __device__ __inline__ uint32_t decode() { return *curPos++; }
};

template <typename T> T tempLoad(Fp* buf, size_t offset);

template <> __device__ __inline__ Fp tempLoad<Fp>(Fp* buf, size_t offset) {
  return buf[offset];
}

template <> __device__ __inline__ FpExt tempLoad<FpExt>(Fp* buf, size_t offset) {
  return FpExt(buf[offset], buf[offset + 1], buf[offset + 2], buf[offset + 3]);
}

template <typename T> __device__ __inline__ void tempStore(Fp* buf, size_t offset, T val);

template <> __device__ __inline__ void tempStore<Fp>(Fp* buf, size_t offset, Fp val) {
  buf[offset] = val;
}
template <> __device__ __inline__ void tempStore<FpExt>(Fp* buf, size_t offset, FpExt val) {
  buf[offset] = val[0];
  buf[offset + 1] = val[1];
  buf[offset + 2] = val[2];
  buf[offset + 3] = val[3];
}

template <typename ConcreteType> struct EncodedProgBase : public EncodedProg {};

__device__ __inline__ constexpr size_t getBackOrZero() { return 0; }
__device__ __inline__ constexpr size_t getBackOrZero(size_t back) { return back; }

#define zllGet(BUF, OFFSET, BACK...) ((BUF)[(OFFSET) * steps + ((cycle - kInvRate * (getBackOrZero(BACK))) & mask)]);
#define zllGetGlobal(BUF, OFFSET) ((BUF)[(OFFSET)])
#define debugIn(X) (X)
#define debugOut(X)                                                                                \
  do {                                                                                             \
  } while (0)

__device__ __inline__ Fp zllConst(int a) {
  return Fp(a);
}

__device__ __inline__ FpExt zllConst(int a, int b, int c, int d) {
  return FpExt(a, b, c, d);
}

__device__ __inline__ FpExt trivialConstraint() {
  return FpExt(0, 0, 0, 0);
}

__device__ __inline__ FpExt zllAndEqz(FpExt inMix, Fp val, size_t mixPowIndex) {
  return inMix + val * poly_mix[mixPowIndex];
}

__device__ __inline__ FpExt zllAndEqz(FpExt inMix, FpExt val, size_t mixPowIndex) {
  return inMix + val * poly_mix[mixPowIndex];
}

__device__ __inline__ FpExt zllAndCond(FpExt inMix, Fp cond, FpExt innerMix, size_t mixPowIndex) {
  return inMix + cond * innerMix * poly_mix[mixPowIndex];
}

__device__ __inline__ FpExt
zllAndCond(FpExt inMix, FpExt cond, FpExt innerMix, size_t mixPowIndex) {
  return inMix + cond * innerMix * poly_mix[mixPowIndex];
}

#include "eval_check_bc.cu.inc"


__global__ void
__launch_bounds__(256, 6)
eval_check(Fp* check,
                           const Fp* ctrl,
                           const Fp* data,
                           const Fp* accum,
                           const Fp* mix,
                           const Fp* out,
                           const Fp rou,
                           uint32_t po2,
                           uint32_t domain) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= domain)
    return;
  FpExt tot = execByteCode<Keccak>(cycle, domain, data, out, poly_mix);
  Fp x = pow(rou, cycle);
  Fp y = pow(Fp(3) * x, 1 << po2);
  FpExt ret = tot * inv(y - Fp(1));
  check[domain * 0 + cycle] = ret[0];
  check[domain * 1 + cycle] = ret[1];
  check[domain * 2 + cycle] = ret[2];
  check[domain * 3 + cycle] = ret[3];
}

} // namespace risc0::circuit::keccak::cuda

extern "C" {

using namespace risc0::circuit::keccak::cuda;

const char* risc0_circuit_keccak_cuda_eval_check(Fp* check,
                                                 const Fp* ctrl,
                                                 const Fp* data,
                                                 const Fp* accum,
                                                 const Fp* mix,
                                                 const Fp* out,
                                                 const Fp& rou,
                                                 uint32_t po2,
                                                 uint32_t domain,
                                                 const FpExt* poly_mix_pows) {
  try {
    CUDA_OK(hipDeviceSynchronize());
    CudaStream stream;
    auto cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

} // extern "C"
