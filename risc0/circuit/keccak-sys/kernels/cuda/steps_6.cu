#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

__device__ NondetRegStruct exec_Xor5(ExecContext& ctx,
                                     Val5Array arg0,
                                     BoundLayout<NondetRegLayout> layout1) {
  // Xor5(zirgen/circuit/keccak2/xor5.zir:7)
  Val x2 = ((arg0[0] + arg0[1]) + arg0[2]);
  Val x3 = ((x2 + arg0[3]) + arg0[4]);
  // Xor5(zirgen/circuit/keccak2/xor5.zir:8)
  NondetRegStruct x4 = exec_NondetBitReg(ctx, bitAnd(x3, Val(1)), layout1);
  // Xor5(zirgen/circuit/keccak2/xor5.zir:9)
  Val x5 = (x3 - x4._super);
  // Xor5(zirgen/circuit/keccak2/xor5.zir:10)
  Val x6 = ((x5 * (Val(2) - x5)) * (Val(4) - x5));
  EQZ(x6, "Xor5(zirgen/circuit/keccak2/xor5.zir:10)");
  return x4;
}
__device__ TopStateStruct exec_ShaNextBlockCycle(ExecContext& ctx,
                                                 TopStateStruct arg0,
                                                 BoundLayout<ShaNextBlockCycleLayout> layout1) {
  // Log(<preamble>:22)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:236)
  INVOKE_EXTERN(ctx, log, "ShaNextBlockCycle", std::initializer_list<Val>{});
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:238)
  Val4Array x2 = Val4Array{Val(0), Val(1), Val(2), Val(3)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  Val x3 = (arg0.bits[225]._super._super * Val(2));
  Val x4 = (arg0.bits[226]._super._super * Val(4));
  Val x5 = (arg0.bits[227]._super._super * Val(8));
  Val x6 = (arg0.bits[228]._super._super * Val(16));
  Val x7 = (arg0.bits[229]._super._super * Val(32));
  Val x8 = (arg0.bits[230]._super._super * Val(64));
  Val x9 = (arg0.bits[231]._super._super * Val(128));
  Val x10 = (arg0.bits[232]._super._super * Val(256));
  Val x11 = (arg0.bits[233]._super._super * Val(512));
  Val x12 = (arg0.bits[234]._super._super * Val(1024));
  Val x13 = (arg0.bits[235]._super._super * Val(2048));
  Val x14 = (arg0.bits[236]._super._super * Val(4096));
  Val x15 = (arg0.bits[237]._super._super * Val(8192));
  Val x16 = (arg0.bits[238]._super._super * Val(16384));
  Val x17 = (arg0.bits[239]._super._super * Val(32768));
  Val x18 = (arg0.bits[224]._super._super + x3);
  Val x19 = (((x18 + x4) + x5) + x6);
  Val x20 = (((x19 + x7) + x8) + x9);
  Val x21 = (((x20 + x10) + x11) + x12);
  Val x22 = (((x21 + x13) + x14) + x15);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x23 =
      Pack_32__16__Super_SuperStruct{._super = ((x22 + x16) + x17)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x24 = (arg0.bits[241]._super._super * Val(2));
  Val x25 = (arg0.bits[242]._super._super * Val(4));
  Val x26 = (arg0.bits[243]._super._super * Val(8));
  Val x27 = (arg0.bits[244]._super._super * Val(16));
  Val x28 = (arg0.bits[245]._super._super * Val(32));
  Val x29 = (arg0.bits[246]._super._super * Val(64));
  Val x30 = (arg0.bits[247]._super._super * Val(128));
  Val x31 = (arg0.bits[248]._super._super * Val(256));
  Val x32 = (arg0.bits[249]._super._super * Val(512));
  Val x33 = (arg0.bits[250]._super._super * Val(1024));
  Val x34 = (arg0.bits[251]._super._super * Val(2048));
  Val x35 = (arg0.bits[252]._super._super * Val(4096));
  Val x36 = (arg0.bits[253]._super._super * Val(8192));
  Val x37 = (arg0.bits[254]._super._super * Val(16384));
  Val x38 = (arg0.bits[255]._super._super * Val(32768));
  Val x39 = (arg0.bits[240]._super._super + x24);
  Val x40 = (((x39 + x25) + x26) + x27);
  Val x41 = (((x40 + x28) + x29) + x30);
  Val x42 = (((x41 + x31) + x32) + x33);
  Val x43 = (((x42 + x34) + x35) + x36);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x44 =
      Pack_32__16__Super_SuperStruct{._super = ((x43 + x37) + x38)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x45 = (arg0.bits[193]._super._super * Val(2));
  Val x46 = (arg0.bits[194]._super._super * Val(4));
  Val x47 = (arg0.bits[195]._super._super * Val(8));
  Val x48 = (arg0.bits[196]._super._super * Val(16));
  Val x49 = (arg0.bits[197]._super._super * Val(32));
  Val x50 = (arg0.bits[198]._super._super * Val(64));
  Val x51 = (arg0.bits[199]._super._super * Val(128));
  Val x52 = (arg0.bits[200]._super._super * Val(256));
  Val x53 = (arg0.bits[201]._super._super * Val(512));
  Val x54 = (arg0.bits[202]._super._super * Val(1024));
  Val x55 = (arg0.bits[203]._super._super * Val(2048));
  Val x56 = (arg0.bits[204]._super._super * Val(4096));
  Val x57 = (arg0.bits[205]._super._super * Val(8192));
  Val x58 = (arg0.bits[206]._super._super * Val(16384));
  Val x59 = (arg0.bits[207]._super._super * Val(32768));
  Val x60 = (arg0.bits[192]._super._super + x45);
  Val x61 = (((x60 + x46) + x47) + x48);
  Val x62 = (((x61 + x49) + x50) + x51);
  Val x63 = (((x62 + x52) + x53) + x54);
  Val x64 = (((x63 + x55) + x56) + x57);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x65 =
      Pack_32__16__Super_SuperStruct{._super = ((x64 + x58) + x59)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x66 = (arg0.bits[209]._super._super * Val(2));
  Val x67 = (arg0.bits[210]._super._super * Val(4));
  Val x68 = (arg0.bits[211]._super._super * Val(8));
  Val x69 = (arg0.bits[212]._super._super * Val(16));
  Val x70 = (arg0.bits[213]._super._super * Val(32));
  Val x71 = (arg0.bits[214]._super._super * Val(64));
  Val x72 = (arg0.bits[215]._super._super * Val(128));
  Val x73 = (arg0.bits[216]._super._super * Val(256));
  Val x74 = (arg0.bits[217]._super._super * Val(512));
  Val x75 = (arg0.bits[218]._super._super * Val(1024));
  Val x76 = (arg0.bits[219]._super._super * Val(2048));
  Val x77 = (arg0.bits[220]._super._super * Val(4096));
  Val x78 = (arg0.bits[221]._super._super * Val(8192));
  Val x79 = (arg0.bits[222]._super._super * Val(16384));
  Val x80 = (arg0.bits[223]._super._super * Val(32768));
  Val x81 = (arg0.bits[208]._super._super + x66);
  Val x82 = (((x81 + x67) + x68) + x69);
  Val x83 = (((x82 + x70) + x71) + x72);
  Val x84 = (((x83 + x73) + x74) + x75);
  Val x85 = (((x84 + x76) + x77) + x78);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x86 =
      Pack_32__16__Super_SuperStruct{._super = ((x85 + x79) + x80)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x87 = (arg0.bits[161]._super._super * Val(2));
  Val x88 = (arg0.bits[162]._super._super * Val(4));
  Val x89 = (arg0.bits[163]._super._super * Val(8));
  Val x90 = (arg0.bits[164]._super._super * Val(16));
  Val x91 = (arg0.bits[165]._super._super * Val(32));
  Val x92 = (arg0.bits[166]._super._super * Val(64));
  Val x93 = (arg0.bits[167]._super._super * Val(128));
  Val x94 = (arg0.bits[168]._super._super * Val(256));
  Val x95 = (arg0.bits[169]._super._super * Val(512));
  Val x96 = (arg0.bits[170]._super._super * Val(1024));
  Val x97 = (arg0.bits[171]._super._super * Val(2048));
  Val x98 = (arg0.bits[172]._super._super * Val(4096));
  Val x99 = (arg0.bits[173]._super._super * Val(8192));
  Val x100 = (arg0.bits[174]._super._super * Val(16384));
  Val x101 = (arg0.bits[175]._super._super * Val(32768));
  Val x102 = (arg0.bits[160]._super._super + x87);
  Val x103 = (((x102 + x88) + x89) + x90);
  Val x104 = (((x103 + x91) + x92) + x93);
  Val x105 = (((x104 + x94) + x95) + x96);
  Val x106 = (((x105 + x97) + x98) + x99);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x107 =
      Pack_32__16__Super_SuperStruct{._super = ((x106 + x100) + x101)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x108 = (arg0.bits[177]._super._super * Val(2));
  Val x109 = (arg0.bits[178]._super._super * Val(4));
  Val x110 = (arg0.bits[179]._super._super * Val(8));
  Val x111 = (arg0.bits[180]._super._super * Val(16));
  Val x112 = (arg0.bits[181]._super._super * Val(32));
  Val x113 = (arg0.bits[182]._super._super * Val(64));
  Val x114 = (arg0.bits[183]._super._super * Val(128));
  Val x115 = (arg0.bits[184]._super._super * Val(256));
  Val x116 = (arg0.bits[185]._super._super * Val(512));
  Val x117 = (arg0.bits[186]._super._super * Val(1024));
  Val x118 = (arg0.bits[187]._super._super * Val(2048));
  Val x119 = (arg0.bits[188]._super._super * Val(4096));
  Val x120 = (arg0.bits[189]._super._super * Val(8192));
  Val x121 = (arg0.bits[190]._super._super * Val(16384));
  Val x122 = (arg0.bits[191]._super._super * Val(32768));
  Val x123 = (arg0.bits[176]._super._super + x108);
  Val x124 = (((x123 + x109) + x110) + x111);
  Val x125 = (((x124 + x112) + x113) + x114);
  Val x126 = (((x125 + x115) + x116) + x117);
  Val x127 = (((x126 + x118) + x119) + x120);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x128 =
      Pack_32__16__Super_SuperStruct{._super = ((x127 + x121) + x122)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x129 = (arg0.bits[129]._super._super * Val(2));
  Val x130 = (arg0.bits[130]._super._super * Val(4));
  Val x131 = (arg0.bits[131]._super._super * Val(8));
  Val x132 = (arg0.bits[132]._super._super * Val(16));
  Val x133 = (arg0.bits[133]._super._super * Val(32));
  Val x134 = (arg0.bits[134]._super._super * Val(64));
  Val x135 = (arg0.bits[135]._super._super * Val(128));
  Val x136 = (arg0.bits[136]._super._super * Val(256));
  Val x137 = (arg0.bits[137]._super._super * Val(512));
  Val x138 = (arg0.bits[138]._super._super * Val(1024));
  Val x139 = (arg0.bits[139]._super._super * Val(2048));
  Val x140 = (arg0.bits[140]._super._super * Val(4096));
  Val x141 = (arg0.bits[141]._super._super * Val(8192));
  Val x142 = (arg0.bits[142]._super._super * Val(16384));
  Val x143 = (arg0.bits[143]._super._super * Val(32768));
  Val x144 = (arg0.bits[128]._super._super + x129);
  Val x145 = (((x144 + x130) + x131) + x132);
  Val x146 = (((x145 + x133) + x134) + x135);
  Val x147 = (((x146 + x136) + x137) + x138);
  Val x148 = (((x147 + x139) + x140) + x141);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x149 =
      Pack_32__16__Super_SuperStruct{._super = ((x148 + x142) + x143)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x150 = (arg0.bits[145]._super._super * Val(2));
  Val x151 = (arg0.bits[146]._super._super * Val(4));
  Val x152 = (arg0.bits[147]._super._super * Val(8));
  Val x153 = (arg0.bits[148]._super._super * Val(16));
  Val x154 = (arg0.bits[149]._super._super * Val(32));
  Val x155 = (arg0.bits[150]._super._super * Val(64));
  Val x156 = (arg0.bits[151]._super._super * Val(128));
  Val x157 = (arg0.bits[152]._super._super * Val(256));
  Val x158 = (arg0.bits[153]._super._super * Val(512));
  Val x159 = (arg0.bits[154]._super._super * Val(1024));
  Val x160 = (arg0.bits[155]._super._super * Val(2048));
  Val x161 = (arg0.bits[156]._super._super * Val(4096));
  Val x162 = (arg0.bits[157]._super._super * Val(8192));
  Val x163 = (arg0.bits[158]._super._super * Val(16384));
  Val x164 = (arg0.bits[159]._super._super * Val(32768));
  Val x165 = (arg0.bits[144]._super._super + x150);
  Val x166 = (((x165 + x151) + x152) + x153);
  Val x167 = (((x166 + x154) + x155) + x156);
  Val x168 = (((x167 + x157) + x158) + x159);
  Val x169 = (((x168 + x160) + x161) + x162);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x170 =
      Pack_32__16__Super_SuperStruct{._super = ((x169 + x163) + x164)};
  Pack_32__16_Struct4Array x171 = Pack_32__16_Struct4Array{
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x23, x44}},
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x65, x86}},
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x107, x128}},
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x149, x170}}};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:239)
  Val x172 = (arg0.bits[481]._super._super * Val(2));
  Val x173 = (arg0.bits[482]._super._super * Val(4));
  Val x174 = (arg0.bits[483]._super._super * Val(8));
  Val x175 = (arg0.bits[484]._super._super * Val(16));
  Val x176 = (arg0.bits[485]._super._super * Val(32));
  Val x177 = (arg0.bits[486]._super._super * Val(64));
  Val x178 = (arg0.bits[487]._super._super * Val(128));
  Val x179 = (arg0.bits[488]._super._super * Val(256));
  Val x180 = (arg0.bits[489]._super._super * Val(512));
  Val x181 = (arg0.bits[490]._super._super * Val(1024));
  Val x182 = (arg0.bits[491]._super._super * Val(2048));
  Val x183 = (arg0.bits[492]._super._super * Val(4096));
  Val x184 = (arg0.bits[493]._super._super * Val(8192));
  Val x185 = (arg0.bits[494]._super._super * Val(16384));
  Val x186 = (arg0.bits[495]._super._super * Val(32768));
  Val x187 = (arg0.bits[480]._super._super + x172);
  Val x188 = (((x187 + x173) + x174) + x175);
  Val x189 = (((x188 + x176) + x177) + x178);
  Val x190 = (((x189 + x179) + x180) + x181);
  Val x191 = (((x190 + x182) + x183) + x184);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x192 =
      Pack_32__16__Super_SuperStruct{._super = ((x191 + x185) + x186)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x193 = (arg0.bits[497]._super._super * Val(2));
  Val x194 = (arg0.bits[498]._super._super * Val(4));
  Val x195 = (arg0.bits[499]._super._super * Val(8));
  Val x196 = (arg0.bits[500]._super._super * Val(16));
  Val x197 = (arg0.bits[501]._super._super * Val(32));
  Val x198 = (arg0.bits[502]._super._super * Val(64));
  Val x199 = (arg0.bits[503]._super._super * Val(128));
  Val x200 = (arg0.bits[504]._super._super * Val(256));
  Val x201 = (arg0.bits[505]._super._super * Val(512));
  Val x202 = (arg0.bits[506]._super._super * Val(1024));
  Val x203 = (arg0.bits[507]._super._super * Val(2048));
  Val x204 = (arg0.bits[508]._super._super * Val(4096));
  Val x205 = (arg0.bits[509]._super._super * Val(8192));
  Val x206 = (arg0.bits[510]._super._super * Val(16384));
  Val x207 = (arg0.bits[511]._super._super * Val(32768));
  Val x208 = (arg0.bits[496]._super._super + x193);
  Val x209 = (((x208 + x194) + x195) + x196);
  Val x210 = (((x209 + x197) + x198) + x199);
  Val x211 = (((x210 + x200) + x201) + x202);
  Val x212 = (((x211 + x203) + x204) + x205);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x213 =
      Pack_32__16__Super_SuperStruct{._super = ((x212 + x206) + x207)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x214 = (arg0.bits[449]._super._super * Val(2));
  Val x215 = (arg0.bits[450]._super._super * Val(4));
  Val x216 = (arg0.bits[451]._super._super * Val(8));
  Val x217 = (arg0.bits[452]._super._super * Val(16));
  Val x218 = (arg0.bits[453]._super._super * Val(32));
  Val x219 = (arg0.bits[454]._super._super * Val(64));
  Val x220 = (arg0.bits[455]._super._super * Val(128));
  Val x221 = (arg0.bits[456]._super._super * Val(256));
  Val x222 = (arg0.bits[457]._super._super * Val(512));
  Val x223 = (arg0.bits[458]._super._super * Val(1024));
  Val x224 = (arg0.bits[459]._super._super * Val(2048));
  Val x225 = (arg0.bits[460]._super._super * Val(4096));
  Val x226 = (arg0.bits[461]._super._super * Val(8192));
  Val x227 = (arg0.bits[462]._super._super * Val(16384));
  Val x228 = (arg0.bits[463]._super._super * Val(32768));
  Val x229 = (arg0.bits[448]._super._super + x214);
  Val x230 = (((x229 + x215) + x216) + x217);
  Val x231 = (((x230 + x218) + x219) + x220);
  Val x232 = (((x231 + x221) + x222) + x223);
  Val x233 = (((x232 + x224) + x225) + x226);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x234 =
      Pack_32__16__Super_SuperStruct{._super = ((x233 + x227) + x228)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x235 = (arg0.bits[465]._super._super * Val(2));
  Val x236 = (arg0.bits[466]._super._super * Val(4));
  Val x237 = (arg0.bits[467]._super._super * Val(8));
  Val x238 = (arg0.bits[468]._super._super * Val(16));
  Val x239 = (arg0.bits[469]._super._super * Val(32));
  Val x240 = (arg0.bits[470]._super._super * Val(64));
  Val x241 = (arg0.bits[471]._super._super * Val(128));
  Val x242 = (arg0.bits[472]._super._super * Val(256));
  Val x243 = (arg0.bits[473]._super._super * Val(512));
  Val x244 = (arg0.bits[474]._super._super * Val(1024));
  Val x245 = (arg0.bits[475]._super._super * Val(2048));
  Val x246 = (arg0.bits[476]._super._super * Val(4096));
  Val x247 = (arg0.bits[477]._super._super * Val(8192));
  Val x248 = (arg0.bits[478]._super._super * Val(16384));
  Val x249 = (arg0.bits[479]._super._super * Val(32768));
  Val x250 = (arg0.bits[464]._super._super + x235);
  Val x251 = (((x250 + x236) + x237) + x238);
  Val x252 = (((x251 + x239) + x240) + x241);
  Val x253 = (((x252 + x242) + x243) + x244);
  Val x254 = (((x253 + x245) + x246) + x247);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x255 =
      Pack_32__16__Super_SuperStruct{._super = ((x254 + x248) + x249)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x256 = (arg0.bits[417]._super._super * Val(2));
  Val x257 = (arg0.bits[418]._super._super * Val(4));
  Val x258 = (arg0.bits[419]._super._super * Val(8));
  Val x259 = (arg0.bits[420]._super._super * Val(16));
  Val x260 = (arg0.bits[421]._super._super * Val(32));
  Val x261 = (arg0.bits[422]._super._super * Val(64));
  Val x262 = (arg0.bits[423]._super._super * Val(128));
  Val x263 = (arg0.bits[424]._super._super * Val(256));
  Val x264 = (arg0.bits[425]._super._super * Val(512));
  Val x265 = (arg0.bits[426]._super._super * Val(1024));
  Val x266 = (arg0.bits[427]._super._super * Val(2048));
  Val x267 = (arg0.bits[428]._super._super * Val(4096));
  Val x268 = (arg0.bits[429]._super._super * Val(8192));
  Val x269 = (arg0.bits[430]._super._super * Val(16384));
  Val x270 = (arg0.bits[431]._super._super * Val(32768));
  Val x271 = (arg0.bits[416]._super._super + x256);
  Val x272 = (((x271 + x257) + x258) + x259);
  Val x273 = (((x272 + x260) + x261) + x262);
  Val x274 = (((x273 + x263) + x264) + x265);
  Val x275 = (((x274 + x266) + x267) + x268);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x276 =
      Pack_32__16__Super_SuperStruct{._super = ((x275 + x269) + x270)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x277 = (arg0.bits[433]._super._super * Val(2));
  Val x278 = (arg0.bits[434]._super._super * Val(4));
  Val x279 = (arg0.bits[435]._super._super * Val(8));
  Val x280 = (arg0.bits[436]._super._super * Val(16));
  Val x281 = (arg0.bits[437]._super._super * Val(32));
  Val x282 = (arg0.bits[438]._super._super * Val(64));
  Val x283 = (arg0.bits[439]._super._super * Val(128));
  Val x284 = (arg0.bits[440]._super._super * Val(256));
  Val x285 = (arg0.bits[441]._super._super * Val(512));
  Val x286 = (arg0.bits[442]._super._super * Val(1024));
  Val x287 = (arg0.bits[443]._super._super * Val(2048));
  Val x288 = (arg0.bits[444]._super._super * Val(4096));
  Val x289 = (arg0.bits[445]._super._super * Val(8192));
  Val x290 = (arg0.bits[446]._super._super * Val(16384));
  Val x291 = (arg0.bits[447]._super._super * Val(32768));
  Val x292 = (arg0.bits[432]._super._super + x277);
  Val x293 = (((x292 + x278) + x279) + x280);
  Val x294 = (((x293 + x281) + x282) + x283);
  Val x295 = (((x294 + x284) + x285) + x286);
  Val x296 = (((x295 + x287) + x288) + x289);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x297 =
      Pack_32__16__Super_SuperStruct{._super = ((x296 + x290) + x291)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x298 = (arg0.bits[385]._super._super * Val(2));
  Val x299 = (arg0.bits[386]._super._super * Val(4));
  Val x300 = (arg0.bits[387]._super._super * Val(8));
  Val x301 = (arg0.bits[388]._super._super * Val(16));
  Val x302 = (arg0.bits[389]._super._super * Val(32));
  Val x303 = (arg0.bits[390]._super._super * Val(64));
  Val x304 = (arg0.bits[391]._super._super * Val(128));
  Val x305 = (arg0.bits[392]._super._super * Val(256));
  Val x306 = (arg0.bits[393]._super._super * Val(512));
  Val x307 = (arg0.bits[394]._super._super * Val(1024));
  Val x308 = (arg0.bits[395]._super._super * Val(2048));
  Val x309 = (arg0.bits[396]._super._super * Val(4096));
  Val x310 = (arg0.bits[397]._super._super * Val(8192));
  Val x311 = (arg0.bits[398]._super._super * Val(16384));
  Val x312 = (arg0.bits[399]._super._super * Val(32768));
  Val x313 = (arg0.bits[384]._super._super + x298);
  Val x314 = (((x313 + x299) + x300) + x301);
  Val x315 = (((x314 + x302) + x303) + x304);
  Val x316 = (((x315 + x305) + x306) + x307);
  Val x317 = (((x316 + x308) + x309) + x310);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x318 =
      Pack_32__16__Super_SuperStruct{._super = ((x317 + x311) + x312)};
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  Val x319 = (arg0.bits[401]._super._super * Val(2));
  Val x320 = (arg0.bits[402]._super._super * Val(4));
  Val x321 = (arg0.bits[403]._super._super * Val(8));
  Val x322 = (arg0.bits[404]._super._super * Val(16));
  Val x323 = (arg0.bits[405]._super._super * Val(32));
  Val x324 = (arg0.bits[406]._super._super * Val(64));
  Val x325 = (arg0.bits[407]._super._super * Val(128));
  Val x326 = (arg0.bits[408]._super._super * Val(256));
  Val x327 = (arg0.bits[409]._super._super * Val(512));
  Val x328 = (arg0.bits[410]._super._super * Val(1024));
  Val x329 = (arg0.bits[411]._super._super * Val(2048));
  Val x330 = (arg0.bits[412]._super._super * Val(4096));
  Val x331 = (arg0.bits[413]._super._super * Val(8192));
  Val x332 = (arg0.bits[414]._super._super * Val(16384));
  Val x333 = (arg0.bits[415]._super._super * Val(32768));
  Val x334 = (arg0.bits[400]._super._super + x319);
  Val x335 = (((x334 + x320) + x321) + x322);
  Val x336 = (((x335 + x323) + x324) + x325);
  Val x337 = (((x336 + x326) + x327) + x328);
  Val x338 = (((x337 + x329) + x330) + x331);
  // Pack(zirgen/circuit/keccak2/pack.zir:31)
  Pack_32__16__Super_SuperStruct x339 =
      Pack_32__16__Super_SuperStruct{._super = ((x338 + x332) + x333)};
  Pack_32__16_Struct4Array x340 = Pack_32__16_Struct4Array{
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x192, x213}},
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x234, x255}},
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x276, x297}},
      Pack_32__16_Struct{._super = Pack_32__16__Super_SuperStruct2Array{x318, x339}}};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:241)
  NondetRegStruct2Array x341 = NondetRegStruct2Array{arg0.sflat[0], arg0.sflat[1]};
  NondetRegStruct2Array x342 = NondetRegStruct2Array{arg0.sflat[2], arg0.sflat[3]};
  NondetRegStruct2Array x343 = NondetRegStruct2Array{arg0.sflat[4], arg0.sflat[5]};
  NondetRegStruct2Array x344 = NondetRegStruct2Array{arg0.sflat[6], arg0.sflat[7]};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:242)
  NondetRegStruct2Array x345 = NondetRegStruct2Array{arg0.sflat[8], arg0.sflat[9]};
  NondetRegStruct2Array x346 = NondetRegStruct2Array{arg0.sflat[10], arg0.sflat[11]};
  NondetRegStruct2Array x347 = NondetRegStruct2Array{arg0.sflat[12], arg0.sflat[13]};
  NondetRegStruct2Array x348 = NondetRegStruct2Array{arg0.sflat[14], arg0.sflat[15]};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:244)
  UnpackReg_32__16_Struct4Array x349 =
      map(x2,
          LAYOUT_LOOKUP(layout1, totA),
          ([&](Val4Array::value_type x350,
               BoundLayout<CarryAndExpandLayout4LayoutArray::value_type> x351) {
            // Add2(zirgen/circuit/keccak2/sha2.zir:30)
            Val x352 = (x171[to_size_t(x350)]._super[0]._super +
                        ShaNextBlockCyclePrevA_SuperStruct4Array{
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x341},
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x342},
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x343},
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x344}}[to_size_t(x350)]
                            ._super[0]
                            ._super);
            Val x353 = (x171[to_size_t(x350)]._super[1]._super +
                        ShaNextBlockCyclePrevA_SuperStruct4Array{
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x341},
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x342},
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x343},
                            ShaNextBlockCyclePrevA_SuperStruct{._super = x344}}[to_size_t(x350)]
                            ._super[1]
                            ._super);
            UnpackReg_32__16_Struct x354 = exec_CarryAndExpand(ctx, Val2Array{x352, x353}, x351);
            return x354;
          }));
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:245)
  UnpackReg_32__16_Struct4Array x355 =
      map(x2,
          LAYOUT_LOOKUP(layout1, totE),
          ([&](Val4Array::value_type x356,
               BoundLayout<CarryAndExpandLayout4LayoutArray::value_type> x357) {
            // Add2(zirgen/circuit/keccak2/sha2.zir:30)
            Val x358 = (x340[to_size_t(x356)]._super[0]._super +
                        ShaNextBlockCyclePrevE_SuperStruct4Array{
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x345},
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x346},
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x347},
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x348}}[to_size_t(x356)]
                            ._super[0]
                            ._super);
            Val x359 = (x340[to_size_t(x356)]._super[1]._super +
                        ShaNextBlockCyclePrevE_SuperStruct4Array{
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x345},
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x346},
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x347},
                            ShaNextBlockCyclePrevE_SuperStruct{._super = x348}}[to_size_t(x356)]
                            ._super[1]
                            ._super);
            UnpackReg_32__16_Struct x360 = exec_CarryAndExpand(ctx, Val2Array{x358, x359}, x357);
            return x360;
          }));
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:257)
  Val x361 = (x349[0]._super[1]._super._super * Val(2));
  Val x362 = (x349[0]._super[2]._super._super * Val(4));
  Val x363 = (x349[0]._super[3]._super._super * Val(8));
  Val x364 = (x349[0]._super[4]._super._super * Val(16));
  Val x365 = (x349[0]._super[5]._super._super * Val(32));
  Val x366 = (x349[0]._super[6]._super._super * Val(64));
  Val x367 = (x349[0]._super[7]._super._super * Val(128));
  Val x368 = (x349[0]._super[8]._super._super * Val(256));
  Val x369 = (x349[0]._super[9]._super._super * Val(512));
  Val x370 = (x349[0]._super[10]._super._super * Val(1024));
  Val x371 = (x349[0]._super[11]._super._super * Val(2048));
  Val x372 = (x349[0]._super[12]._super._super * Val(4096));
  Val x373 = (x349[0]._super[13]._super._super * Val(8192));
  Val x374 = (x349[0]._super[14]._super._super * Val(16384));
  Val x375 = (x349[0]._super[15]._super._super * Val(32768));
  Val x376 = (x349[0]._super[0]._super._super + x361);
  Val x377 = (((x376 + x362) + x363) + x364);
  Val x378 = (((x377 + x365) + x366) + x367);
  Val x379 = (((x378 + x368) + x369) + x370);
  Val x380 = (((x379 + x371) + x372) + x373);
  Val x381 = (x349[0]._super[17]._super._super * Val(2));
  Val x382 = (x349[0]._super[18]._super._super * Val(4));
  Val x383 = (x349[0]._super[19]._super._super * Val(8));
  Val x384 = (x349[0]._super[20]._super._super * Val(16));
  Val x385 = (x349[0]._super[21]._super._super * Val(32));
  Val x386 = (x349[0]._super[22]._super._super * Val(64));
  Val x387 = (x349[0]._super[23]._super._super * Val(128));
  Val x388 = (x349[0]._super[24]._super._super * Val(256));
  Val x389 = (x349[0]._super[25]._super._super * Val(512));
  Val x390 = (x349[0]._super[26]._super._super * Val(1024));
  Val x391 = (x349[0]._super[27]._super._super * Val(2048));
  Val x392 = (x349[0]._super[28]._super._super * Val(4096));
  Val x393 = (x349[0]._super[29]._super._super * Val(8192));
  Val x394 = (x349[0]._super[30]._super._super * Val(16384));
  Val x395 = (x349[0]._super[31]._super._super * Val(32768));
  Val x396 = (x349[0]._super[16]._super._super + x381);
  Val x397 = (((x396 + x382) + x383) + x384);
  Val x398 = (((x397 + x385) + x386) + x387);
  Val x399 = (((x398 + x388) + x389) + x390);
  Val x400 = (((x399 + x391) + x392) + x393);
  Val x401 = (x349[1]._super[1]._super._super * Val(2));
  Val x402 = (x349[1]._super[2]._super._super * Val(4));
  Val x403 = (x349[1]._super[3]._super._super * Val(8));
  Val x404 = (x349[1]._super[4]._super._super * Val(16));
  Val x405 = (x349[1]._super[5]._super._super * Val(32));
  Val x406 = (x349[1]._super[6]._super._super * Val(64));
  Val x407 = (x349[1]._super[7]._super._super * Val(128));
  Val x408 = (x349[1]._super[8]._super._super * Val(256));
  Val x409 = (x349[1]._super[9]._super._super * Val(512));
  Val x410 = (x349[1]._super[10]._super._super * Val(1024));
  Val x411 = (x349[1]._super[11]._super._super * Val(2048));
  Val x412 = (x349[1]._super[12]._super._super * Val(4096));
  Val x413 = (x349[1]._super[13]._super._super * Val(8192));
  Val x414 = (x349[1]._super[14]._super._super * Val(16384));
  Val x415 = (x349[1]._super[15]._super._super * Val(32768));
  Val x416 = (x349[1]._super[0]._super._super + x401);
  Val x417 = (((x416 + x402) + x403) + x404);
  Val x418 = (((x417 + x405) + x406) + x407);
  Val x419 = (((x418 + x408) + x409) + x410);
  Val x420 = (((x419 + x411) + x412) + x413);
  Val x421 = (x349[1]._super[17]._super._super * Val(2));
  Val x422 = (x349[1]._super[18]._super._super * Val(4));
  Val x423 = (x349[1]._super[19]._super._super * Val(8));
  Val x424 = (x349[1]._super[20]._super._super * Val(16));
  Val x425 = (x349[1]._super[21]._super._super * Val(32));
  Val x426 = (x349[1]._super[22]._super._super * Val(64));
  Val x427 = (x349[1]._super[23]._super._super * Val(128));
  Val x428 = (x349[1]._super[24]._super._super * Val(256));
  Val x429 = (x349[1]._super[25]._super._super * Val(512));
  Val x430 = (x349[1]._super[26]._super._super * Val(1024));
  Val x431 = (x349[1]._super[27]._super._super * Val(2048));
  Val x432 = (x349[1]._super[28]._super._super * Val(4096));
  Val x433 = (x349[1]._super[29]._super._super * Val(8192));
  Val x434 = (x349[1]._super[30]._super._super * Val(16384));
  Val x435 = (x349[1]._super[31]._super._super * Val(32768));
  Val x436 = (x349[1]._super[16]._super._super + x421);
  Val x437 = (((x436 + x422) + x423) + x424);
  Val x438 = (((x437 + x425) + x426) + x427);
  Val x439 = (((x438 + x428) + x429) + x430);
  Val x440 = (((x439 + x431) + x432) + x433);
  Val x441 = (x349[2]._super[1]._super._super * Val(2));
  Val x442 = (x349[2]._super[2]._super._super * Val(4));
  Val x443 = (x349[2]._super[3]._super._super * Val(8));
  Val x444 = (x349[2]._super[4]._super._super * Val(16));
  Val x445 = (x349[2]._super[5]._super._super * Val(32));
  Val x446 = (x349[2]._super[6]._super._super * Val(64));
  Val x447 = (x349[2]._super[7]._super._super * Val(128));
  Val x448 = (x349[2]._super[8]._super._super * Val(256));
  Val x449 = (x349[2]._super[9]._super._super * Val(512));
  Val x450 = (x349[2]._super[10]._super._super * Val(1024));
  Val x451 = (x349[2]._super[11]._super._super * Val(2048));
  Val x452 = (x349[2]._super[12]._super._super * Val(4096));
  Val x453 = (x349[2]._super[13]._super._super * Val(8192));
  Val x454 = (x349[2]._super[14]._super._super * Val(16384));
  Val x455 = (x349[2]._super[15]._super._super * Val(32768));
  Val x456 = (x349[2]._super[0]._super._super + x441);
  Val x457 = (((x456 + x442) + x443) + x444);
  Val x458 = (((x457 + x445) + x446) + x447);
  Val x459 = (((x458 + x448) + x449) + x450);
  Val x460 = (((x459 + x451) + x452) + x453);
  Val x461 = (x349[2]._super[17]._super._super * Val(2));
  Val x462 = (x349[2]._super[18]._super._super * Val(4));
  Val x463 = (x349[2]._super[19]._super._super * Val(8));
  Val x464 = (x349[2]._super[20]._super._super * Val(16));
  Val x465 = (x349[2]._super[21]._super._super * Val(32));
  Val x466 = (x349[2]._super[22]._super._super * Val(64));
  Val x467 = (x349[2]._super[23]._super._super * Val(128));
  Val x468 = (x349[2]._super[24]._super._super * Val(256));
  Val x469 = (x349[2]._super[25]._super._super * Val(512));
  Val x470 = (x349[2]._super[26]._super._super * Val(1024));
  Val x471 = (x349[2]._super[27]._super._super * Val(2048));
  Val x472 = (x349[2]._super[28]._super._super * Val(4096));
  Val x473 = (x349[2]._super[29]._super._super * Val(8192));
  Val x474 = (x349[2]._super[30]._super._super * Val(16384));
  Val x475 = (x349[2]._super[31]._super._super * Val(32768));
  Val x476 = (x349[2]._super[16]._super._super + x461);
  Val x477 = (((x476 + x462) + x463) + x464);
  Val x478 = (((x477 + x465) + x466) + x467);
  Val x479 = (((x478 + x468) + x469) + x470);
  Val x480 = (((x479 + x471) + x472) + x473);
  Val x481 = (x349[3]._super[1]._super._super * Val(2));
  Val x482 = (x349[3]._super[2]._super._super * Val(4));
  Val x483 = (x349[3]._super[3]._super._super * Val(8));
  Val x484 = (x349[3]._super[4]._super._super * Val(16));
  Val x485 = (x349[3]._super[5]._super._super * Val(32));
  Val x486 = (x349[3]._super[6]._super._super * Val(64));
  Val x487 = (x349[3]._super[7]._super._super * Val(128));
  Val x488 = (x349[3]._super[8]._super._super * Val(256));
  Val x489 = (x349[3]._super[9]._super._super * Val(512));
  Val x490 = (x349[3]._super[10]._super._super * Val(1024));
  Val x491 = (x349[3]._super[11]._super._super * Val(2048));
  Val x492 = (x349[3]._super[12]._super._super * Val(4096));
  Val x493 = (x349[3]._super[13]._super._super * Val(8192));
  Val x494 = (x349[3]._super[14]._super._super * Val(16384));
  Val x495 = (x349[3]._super[15]._super._super * Val(32768));
  Val x496 = (x349[3]._super[0]._super._super + x481);
  Val x497 = (((x496 + x482) + x483) + x484);
  Val x498 = (((x497 + x485) + x486) + x487);
  Val x499 = (((x498 + x488) + x489) + x490);
  Val x500 = (((x499 + x491) + x492) + x493);
  Val x501 = (x349[3]._super[17]._super._super * Val(2));
  Val x502 = (x349[3]._super[18]._super._super * Val(4));
  Val x503 = (x349[3]._super[19]._super._super * Val(8));
  Val x504 = (x349[3]._super[20]._super._super * Val(16));
  Val x505 = (x349[3]._super[21]._super._super * Val(32));
  Val x506 = (x349[3]._super[22]._super._super * Val(64));
  Val x507 = (x349[3]._super[23]._super._super * Val(128));
  Val x508 = (x349[3]._super[24]._super._super * Val(256));
  Val x509 = (x349[3]._super[25]._super._super * Val(512));
  Val x510 = (x349[3]._super[26]._super._super * Val(1024));
  Val x511 = (x349[3]._super[27]._super._super * Val(2048));
  Val x512 = (x349[3]._super[28]._super._super * Val(4096));
  Val x513 = (x349[3]._super[29]._super._super * Val(8192));
  Val x514 = (x349[3]._super[30]._super._super * Val(16384));
  Val x515 = (x349[3]._super[31]._super._super * Val(32768));
  Val x516 = (x349[3]._super[16]._super._super + x501);
  Val x517 = (((x516 + x502) + x503) + x504);
  Val x518 = (((x517 + x505) + x506) + x507);
  Val x519 = (((x518 + x508) + x509) + x510);
  Val x520 = (((x519 + x511) + x512) + x513);
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:259)
  Val x521 = (x355[0]._super[1]._super._super * Val(2));
  Val x522 = (x355[0]._super[2]._super._super * Val(4));
  Val x523 = (x355[0]._super[3]._super._super * Val(8));
  Val x524 = (x355[0]._super[4]._super._super * Val(16));
  Val x525 = (x355[0]._super[5]._super._super * Val(32));
  Val x526 = (x355[0]._super[6]._super._super * Val(64));
  Val x527 = (x355[0]._super[7]._super._super * Val(128));
  Val x528 = (x355[0]._super[8]._super._super * Val(256));
  Val x529 = (x355[0]._super[9]._super._super * Val(512));
  Val x530 = (x355[0]._super[10]._super._super * Val(1024));
  Val x531 = (x355[0]._super[11]._super._super * Val(2048));
  Val x532 = (x355[0]._super[12]._super._super * Val(4096));
  Val x533 = (x355[0]._super[13]._super._super * Val(8192));
  Val x534 = (x355[0]._super[14]._super._super * Val(16384));
  Val x535 = (x355[0]._super[15]._super._super * Val(32768));
  Val x536 = (x355[0]._super[0]._super._super + x521);
  Val x537 = (((x536 + x522) + x523) + x524);
  Val x538 = (((x537 + x525) + x526) + x527);
  Val x539 = (((x538 + x528) + x529) + x530);
  Val x540 = (((x539 + x531) + x532) + x533);
  Val x541 = (x355[0]._super[17]._super._super * Val(2));
  Val x542 = (x355[0]._super[18]._super._super * Val(4));
  Val x543 = (x355[0]._super[19]._super._super * Val(8));
  Val x544 = (x355[0]._super[20]._super._super * Val(16));
  Val x545 = (x355[0]._super[21]._super._super * Val(32));
  Val x546 = (x355[0]._super[22]._super._super * Val(64));
  Val x547 = (x355[0]._super[23]._super._super * Val(128));
  Val x548 = (x355[0]._super[24]._super._super * Val(256));
  Val x549 = (x355[0]._super[25]._super._super * Val(512));
  Val x550 = (x355[0]._super[26]._super._super * Val(1024));
  Val x551 = (x355[0]._super[27]._super._super * Val(2048));
  Val x552 = (x355[0]._super[28]._super._super * Val(4096));
  Val x553 = (x355[0]._super[29]._super._super * Val(8192));
  Val x554 = (x355[0]._super[30]._super._super * Val(16384));
  Val x555 = (x355[0]._super[31]._super._super * Val(32768));
  Val x556 = (x355[0]._super[16]._super._super + x541);
  Val x557 = (((x556 + x542) + x543) + x544);
  Val x558 = (((x557 + x545) + x546) + x547);
  Val x559 = (((x558 + x548) + x549) + x550);
  Val x560 = (((x559 + x551) + x552) + x553);
  Val x561 = (x355[1]._super[1]._super._super * Val(2));
  Val x562 = (x355[1]._super[2]._super._super * Val(4));
  Val x563 = (x355[1]._super[3]._super._super * Val(8));
  Val x564 = (x355[1]._super[4]._super._super * Val(16));
  Val x565 = (x355[1]._super[5]._super._super * Val(32));
  Val x566 = (x355[1]._super[6]._super._super * Val(64));
  Val x567 = (x355[1]._super[7]._super._super * Val(128));
  Val x568 = (x355[1]._super[8]._super._super * Val(256));
  Val x569 = (x355[1]._super[9]._super._super * Val(512));
  Val x570 = (x355[1]._super[10]._super._super * Val(1024));
  Val x571 = (x355[1]._super[11]._super._super * Val(2048));
  Val x572 = (x355[1]._super[12]._super._super * Val(4096));
  Val x573 = (x355[1]._super[13]._super._super * Val(8192));
  Val x574 = (x355[1]._super[14]._super._super * Val(16384));
  Val x575 = (x355[1]._super[15]._super._super * Val(32768));
  Val x576 = (x355[1]._super[0]._super._super + x561);
  Val x577 = (((x576 + x562) + x563) + x564);
  Val x578 = (((x577 + x565) + x566) + x567);
  Val x579 = (((x578 + x568) + x569) + x570);
  Val x580 = (((x579 + x571) + x572) + x573);
  Val x581 = (x355[1]._super[17]._super._super * Val(2));
  Val x582 = (x355[1]._super[18]._super._super * Val(4));
  Val x583 = (x355[1]._super[19]._super._super * Val(8));
  Val x584 = (x355[1]._super[20]._super._super * Val(16));
  Val x585 = (x355[1]._super[21]._super._super * Val(32));
  Val x586 = (x355[1]._super[22]._super._super * Val(64));
  Val x587 = (x355[1]._super[23]._super._super * Val(128));
  Val x588 = (x355[1]._super[24]._super._super * Val(256));
  Val x589 = (x355[1]._super[25]._super._super * Val(512));
  Val x590 = (x355[1]._super[26]._super._super * Val(1024));
  Val x591 = (x355[1]._super[27]._super._super * Val(2048));
  Val x592 = (x355[1]._super[28]._super._super * Val(4096));
  Val x593 = (x355[1]._super[29]._super._super * Val(8192));
  Val x594 = (x355[1]._super[30]._super._super * Val(16384));
  Val x595 = (x355[1]._super[31]._super._super * Val(32768));
  Val x596 = (x355[1]._super[16]._super._super + x581);
  Val x597 = (((x596 + x582) + x583) + x584);
  Val x598 = (((x597 + x585) + x586) + x587);
  Val x599 = (((x598 + x588) + x589) + x590);
  Val x600 = (((x599 + x591) + x592) + x593);
  Val x601 = (x355[2]._super[1]._super._super * Val(2));
  Val x602 = (x355[2]._super[2]._super._super * Val(4));
  Val x603 = (x355[2]._super[3]._super._super * Val(8));
  Val x604 = (x355[2]._super[4]._super._super * Val(16));
  Val x605 = (x355[2]._super[5]._super._super * Val(32));
  Val x606 = (x355[2]._super[6]._super._super * Val(64));
  Val x607 = (x355[2]._super[7]._super._super * Val(128));
  Val x608 = (x355[2]._super[8]._super._super * Val(256));
  Val x609 = (x355[2]._super[9]._super._super * Val(512));
  Val x610 = (x355[2]._super[10]._super._super * Val(1024));
  Val x611 = (x355[2]._super[11]._super._super * Val(2048));
  Val x612 = (x355[2]._super[12]._super._super * Val(4096));
  Val x613 = (x355[2]._super[13]._super._super * Val(8192));
  Val x614 = (x355[2]._super[14]._super._super * Val(16384));
  Val x615 = (x355[2]._super[15]._super._super * Val(32768));
  Val x616 = (x355[2]._super[0]._super._super + x601);
  Val x617 = (((x616 + x602) + x603) + x604);
  Val x618 = (((x617 + x605) + x606) + x607);
  Val x619 = (((x618 + x608) + x609) + x610);
  Val x620 = (((x619 + x611) + x612) + x613);
  Val x621 = (x355[2]._super[17]._super._super * Val(2));
  Val x622 = (x355[2]._super[18]._super._super * Val(4));
  Val x623 = (x355[2]._super[19]._super._super * Val(8));
  Val x624 = (x355[2]._super[20]._super._super * Val(16));
  Val x625 = (x355[2]._super[21]._super._super * Val(32));
  Val x626 = (x355[2]._super[22]._super._super * Val(64));
  Val x627 = (x355[2]._super[23]._super._super * Val(128));
  Val x628 = (x355[2]._super[24]._super._super * Val(256));
  Val x629 = (x355[2]._super[25]._super._super * Val(512));
  Val x630 = (x355[2]._super[26]._super._super * Val(1024));
  Val x631 = (x355[2]._super[27]._super._super * Val(2048));
  Val x632 = (x355[2]._super[28]._super._super * Val(4096));
  Val x633 = (x355[2]._super[29]._super._super * Val(8192));
  Val x634 = (x355[2]._super[30]._super._super * Val(16384));
  Val x635 = (x355[2]._super[31]._super._super * Val(32768));
  Val x636 = (x355[2]._super[16]._super._super + x621);
  Val x637 = (((x636 + x622) + x623) + x624);
  Val x638 = (((x637 + x625) + x626) + x627);
  Val x639 = (((x638 + x628) + x629) + x630);
  Val x640 = (((x639 + x631) + x632) + x633);
  Val x641 = (x355[3]._super[1]._super._super * Val(2));
  Val x642 = (x355[3]._super[2]._super._super * Val(4));
  Val x643 = (x355[3]._super[3]._super._super * Val(8));
  Val x644 = (x355[3]._super[4]._super._super * Val(16));
  Val x645 = (x355[3]._super[5]._super._super * Val(32));
  Val x646 = (x355[3]._super[6]._super._super * Val(64));
  Val x647 = (x355[3]._super[7]._super._super * Val(128));
  Val x648 = (x355[3]._super[8]._super._super * Val(256));
  Val x649 = (x355[3]._super[9]._super._super * Val(512));
  Val x650 = (x355[3]._super[10]._super._super * Val(1024));
  Val x651 = (x355[3]._super[11]._super._super * Val(2048));
  Val x652 = (x355[3]._super[12]._super._super * Val(4096));
  Val x653 = (x355[3]._super[13]._super._super * Val(8192));
  Val x654 = (x355[3]._super[14]._super._super * Val(16384));
  Val x655 = (x355[3]._super[15]._super._super * Val(32768));
  Val x656 = (x355[3]._super[0]._super._super + x641);
  Val x657 = (((x656 + x642) + x643) + x644);
  Val x658 = (((x657 + x645) + x646) + x647);
  Val x659 = (((x658 + x648) + x649) + x650);
  Val x660 = (((x659 + x651) + x652) + x653);
  Val x661 = (x355[3]._super[17]._super._super * Val(2));
  Val x662 = (x355[3]._super[18]._super._super * Val(4));
  Val x663 = (x355[3]._super[19]._super._super * Val(8));
  Val x664 = (x355[3]._super[20]._super._super * Val(16));
  Val x665 = (x355[3]._super[21]._super._super * Val(32));
  Val x666 = (x355[3]._super[22]._super._super * Val(64));
  Val x667 = (x355[3]._super[23]._super._super * Val(128));
  Val x668 = (x355[3]._super[24]._super._super * Val(256));
  Val x669 = (x355[3]._super[25]._super._super * Val(512));
  Val x670 = (x355[3]._super[26]._super._super * Val(1024));
  Val x671 = (x355[3]._super[27]._super._super * Val(2048));
  Val x672 = (x355[3]._super[28]._super._super * Val(4096));
  Val x673 = (x355[3]._super[29]._super._super * Val(8192));
  Val x674 = (x355[3]._super[30]._super._super * Val(16384));
  Val x675 = (x355[3]._super[31]._super._super * Val(32768));
  Val x676 = (x355[3]._super[16]._super._super + x661);
  Val x677 = (((x676 + x662) + x663) + x664);
  Val x678 = (((x677 + x665) + x666) + x667);
  Val x679 = (((x678 + x668) + x669) + x670);
  Val x680 = (((x679 + x671) + x672) + x673);
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:247)
  Val800Array x681 = Val800Array{Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 x349[3]._super[0]._super._super,
                                 x349[3]._super[1]._super._super,
                                 x349[3]._super[2]._super._super,
                                 x349[3]._super[3]._super._super,
                                 x349[3]._super[4]._super._super,
                                 x349[3]._super[5]._super._super,
                                 x349[3]._super[6]._super._super,
                                 x349[3]._super[7]._super._super,
                                 x349[3]._super[8]._super._super,
                                 x349[3]._super[9]._super._super,
                                 x349[3]._super[10]._super._super,
                                 x349[3]._super[11]._super._super,
                                 x349[3]._super[12]._super._super,
                                 x349[3]._super[13]._super._super,
                                 x349[3]._super[14]._super._super,
                                 x349[3]._super[15]._super._super,
                                 x349[3]._super[16]._super._super,
                                 x349[3]._super[17]._super._super,
                                 x349[3]._super[18]._super._super,
                                 x349[3]._super[19]._super._super,
                                 x349[3]._super[20]._super._super,
                                 x349[3]._super[21]._super._super,
                                 x349[3]._super[22]._super._super,
                                 x349[3]._super[23]._super._super,
                                 x349[3]._super[24]._super._super,
                                 x349[3]._super[25]._super._super,
                                 x349[3]._super[26]._super._super,
                                 x349[3]._super[27]._super._super,
                                 x349[3]._super[28]._super._super,
                                 x349[3]._super[29]._super._super,
                                 x349[3]._super[30]._super._super,
                                 x349[3]._super[31]._super._super,
                                 x349[2]._super[0]._super._super,
                                 x349[2]._super[1]._super._super,
                                 x349[2]._super[2]._super._super,
                                 x349[2]._super[3]._super._super,
                                 x349[2]._super[4]._super._super,
                                 x349[2]._super[5]._super._super,
                                 x349[2]._super[6]._super._super,
                                 x349[2]._super[7]._super._super,
                                 x349[2]._super[8]._super._super,
                                 x349[2]._super[9]._super._super,
                                 x349[2]._super[10]._super._super,
                                 x349[2]._super[11]._super._super,
                                 x349[2]._super[12]._super._super,
                                 x349[2]._super[13]._super._super,
                                 x349[2]._super[14]._super._super,
                                 x349[2]._super[15]._super._super,
                                 x349[2]._super[16]._super._super,
                                 x349[2]._super[17]._super._super,
                                 x349[2]._super[18]._super._super,
                                 x349[2]._super[19]._super._super,
                                 x349[2]._super[20]._super._super,
                                 x349[2]._super[21]._super._super,
                                 x349[2]._super[22]._super._super,
                                 x349[2]._super[23]._super._super,
                                 x349[2]._super[24]._super._super,
                                 x349[2]._super[25]._super._super,
                                 x349[2]._super[26]._super._super,
                                 x349[2]._super[27]._super._super,
                                 x349[2]._super[28]._super._super,
                                 x349[2]._super[29]._super._super,
                                 x349[2]._super[30]._super._super,
                                 x349[2]._super[31]._super._super,
                                 x349[1]._super[0]._super._super,
                                 x349[1]._super[1]._super._super,
                                 x349[1]._super[2]._super._super,
                                 x349[1]._super[3]._super._super,
                                 x349[1]._super[4]._super._super,
                                 x349[1]._super[5]._super._super,
                                 x349[1]._super[6]._super._super,
                                 x349[1]._super[7]._super._super,
                                 x349[1]._super[8]._super._super,
                                 x349[1]._super[9]._super._super,
                                 x349[1]._super[10]._super._super,
                                 x349[1]._super[11]._super._super,
                                 x349[1]._super[12]._super._super,
                                 x349[1]._super[13]._super._super,
                                 x349[1]._super[14]._super._super,
                                 x349[1]._super[15]._super._super,
                                 x349[1]._super[16]._super._super,
                                 x349[1]._super[17]._super._super,
                                 x349[1]._super[18]._super._super,
                                 x349[1]._super[19]._super._super,
                                 x349[1]._super[20]._super._super,
                                 x349[1]._super[21]._super._super,
                                 x349[1]._super[22]._super._super,
                                 x349[1]._super[23]._super._super,
                                 x349[1]._super[24]._super._super,
                                 x349[1]._super[25]._super._super,
                                 x349[1]._super[26]._super._super,
                                 x349[1]._super[27]._super._super,
                                 x349[1]._super[28]._super._super,
                                 x349[1]._super[29]._super._super,
                                 x349[1]._super[30]._super._super,
                                 x349[1]._super[31]._super._super,
                                 x349[0]._super[0]._super._super,
                                 x349[0]._super[1]._super._super,
                                 x349[0]._super[2]._super._super,
                                 x349[0]._super[3]._super._super,
                                 x349[0]._super[4]._super._super,
                                 x349[0]._super[5]._super._super,
                                 x349[0]._super[6]._super._super,
                                 x349[0]._super[7]._super._super,
                                 x349[0]._super[8]._super._super,
                                 x349[0]._super[9]._super._super,
                                 x349[0]._super[10]._super._super,
                                 x349[0]._super[11]._super._super,
                                 x349[0]._super[12]._super._super,
                                 x349[0]._super[13]._super._super,
                                 x349[0]._super[14]._super._super,
                                 x349[0]._super[15]._super._super,
                                 x349[0]._super[16]._super._super,
                                 x349[0]._super[17]._super._super,
                                 x349[0]._super[18]._super._super,
                                 x349[0]._super[19]._super._super,
                                 x349[0]._super[20]._super._super,
                                 x349[0]._super[21]._super._super,
                                 x349[0]._super[22]._super._super,
                                 x349[0]._super[23]._super._super,
                                 x349[0]._super[24]._super._super,
                                 x349[0]._super[25]._super._super,
                                 x349[0]._super[26]._super._super,
                                 x349[0]._super[27]._super._super,
                                 x349[0]._super[28]._super._super,
                                 x349[0]._super[29]._super._super,
                                 x349[0]._super[30]._super._super,
                                 x349[0]._super[31]._super._super,
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 x355[3]._super[0]._super._super,
                                 x355[3]._super[1]._super._super,
                                 x355[3]._super[2]._super._super,
                                 x355[3]._super[3]._super._super,
                                 x355[3]._super[4]._super._super,
                                 x355[3]._super[5]._super._super,
                                 x355[3]._super[6]._super._super,
                                 x355[3]._super[7]._super._super,
                                 x355[3]._super[8]._super._super,
                                 x355[3]._super[9]._super._super,
                                 x355[3]._super[10]._super._super,
                                 x355[3]._super[11]._super._super,
                                 x355[3]._super[12]._super._super,
                                 x355[3]._super[13]._super._super,
                                 x355[3]._super[14]._super._super,
                                 x355[3]._super[15]._super._super,
                                 x355[3]._super[16]._super._super,
                                 x355[3]._super[17]._super._super,
                                 x355[3]._super[18]._super._super,
                                 x355[3]._super[19]._super._super,
                                 x355[3]._super[20]._super._super,
                                 x355[3]._super[21]._super._super,
                                 x355[3]._super[22]._super._super,
                                 x355[3]._super[23]._super._super,
                                 x355[3]._super[24]._super._super,
                                 x355[3]._super[25]._super._super,
                                 x355[3]._super[26]._super._super,
                                 x355[3]._super[27]._super._super,
                                 x355[3]._super[28]._super._super,
                                 x355[3]._super[29]._super._super,
                                 x355[3]._super[30]._super._super,
                                 x355[3]._super[31]._super._super,
                                 x355[2]._super[0]._super._super,
                                 x355[2]._super[1]._super._super,
                                 x355[2]._super[2]._super._super,
                                 x355[2]._super[3]._super._super,
                                 x355[2]._super[4]._super._super,
                                 x355[2]._super[5]._super._super,
                                 x355[2]._super[6]._super._super,
                                 x355[2]._super[7]._super._super,
                                 x355[2]._super[8]._super._super,
                                 x355[2]._super[9]._super._super,
                                 x355[2]._super[10]._super._super,
                                 x355[2]._super[11]._super._super,
                                 x355[2]._super[12]._super._super,
                                 x355[2]._super[13]._super._super,
                                 x355[2]._super[14]._super._super,
                                 x355[2]._super[15]._super._super,
                                 x355[2]._super[16]._super._super,
                                 x355[2]._super[17]._super._super,
                                 x355[2]._super[18]._super._super,
                                 x355[2]._super[19]._super._super,
                                 x355[2]._super[20]._super._super,
                                 x355[2]._super[21]._super._super,
                                 x355[2]._super[22]._super._super,
                                 x355[2]._super[23]._super._super,
                                 x355[2]._super[24]._super._super,
                                 x355[2]._super[25]._super._super,
                                 x355[2]._super[26]._super._super,
                                 x355[2]._super[27]._super._super,
                                 x355[2]._super[28]._super._super,
                                 x355[2]._super[29]._super._super,
                                 x355[2]._super[30]._super._super,
                                 x355[2]._super[31]._super._super,
                                 x355[1]._super[0]._super._super,
                                 x355[1]._super[1]._super._super,
                                 x355[1]._super[2]._super._super,
                                 x355[1]._super[3]._super._super,
                                 x355[1]._super[4]._super._super,
                                 x355[1]._super[5]._super._super,
                                 x355[1]._super[6]._super._super,
                                 x355[1]._super[7]._super._super,
                                 x355[1]._super[8]._super._super,
                                 x355[1]._super[9]._super._super,
                                 x355[1]._super[10]._super._super,
                                 x355[1]._super[11]._super._super,
                                 x355[1]._super[12]._super._super,
                                 x355[1]._super[13]._super._super,
                                 x355[1]._super[14]._super._super,
                                 x355[1]._super[15]._super._super,
                                 x355[1]._super[16]._super._super,
                                 x355[1]._super[17]._super._super,
                                 x355[1]._super[18]._super._super,
                                 x355[1]._super[19]._super._super,
                                 x355[1]._super[20]._super._super,
                                 x355[1]._super[21]._super._super,
                                 x355[1]._super[22]._super._super,
                                 x355[1]._super[23]._super._super,
                                 x355[1]._super[24]._super._super,
                                 x355[1]._super[25]._super._super,
                                 x355[1]._super[26]._super._super,
                                 x355[1]._super[27]._super._super,
                                 x355[1]._super[28]._super._super,
                                 x355[1]._super[29]._super._super,
                                 x355[1]._super[30]._super._super,
                                 x355[1]._super[31]._super._super,
                                 x355[0]._super[0]._super._super,
                                 x355[0]._super[1]._super._super,
                                 x355[0]._super[2]._super._super,
                                 x355[0]._super[3]._super._super,
                                 x355[0]._super[4]._super._super,
                                 x355[0]._super[5]._super._super,
                                 x355[0]._super[6]._super._super,
                                 x355[0]._super[7]._super._super,
                                 x355[0]._super[8]._super._super,
                                 x355[0]._super[9]._super._super,
                                 x355[0]._super[10]._super._super,
                                 x355[0]._super[11]._super._super,
                                 x355[0]._super[12]._super._super,
                                 x355[0]._super[13]._super._super,
                                 x355[0]._super[14]._super._super,
                                 x355[0]._super[15]._super._super,
                                 x355[0]._super[16]._super._super,
                                 x355[0]._super[17]._super._super,
                                 x355[0]._super[18]._super._super,
                                 x355[0]._super[19]._super._super,
                                 x355[0]._super[20]._super._super,
                                 x355[0]._super[21]._super._super,
                                 x355[0]._super[22]._super._super,
                                 x355[0]._super[23]._super._super,
                                 x355[0]._super[24]._super._super,
                                 x355[0]._super[25]._super._super,
                                 x355[0]._super[26]._super._super,
                                 x355[0]._super[27]._super._super,
                                 x355[0]._super[28]._super._super,
                                 x355[0]._super[29]._super._super,
                                 x355[0]._super[30]._super._super,
                                 x355[0]._super[31]._super._super,
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0),
                                 Val(0)};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:235)
  Val100Array x682 = Val100Array{
      arg0.kflat[0]._super._super,  arg0.kflat[1]._super._super,  arg0.kflat[2]._super._super,
      arg0.kflat[3]._super._super,  arg0.kflat[4]._super._super,  arg0.kflat[5]._super._super,
      arg0.kflat[6]._super._super,  arg0.kflat[7]._super._super,  arg0.kflat[8]._super._super,
      arg0.kflat[9]._super._super,  arg0.kflat[10]._super._super, arg0.kflat[11]._super._super,
      arg0.kflat[12]._super._super, arg0.kflat[13]._super._super, arg0.kflat[14]._super._super,
      arg0.kflat[15]._super._super, arg0.kflat[16]._super._super, arg0.kflat[17]._super._super,
      arg0.kflat[18]._super._super, arg0.kflat[19]._super._super, arg0.kflat[20]._super._super,
      arg0.kflat[21]._super._super, arg0.kflat[22]._super._super, arg0.kflat[23]._super._super,
      arg0.kflat[24]._super._super, arg0.kflat[25]._super._super, arg0.kflat[26]._super._super,
      arg0.kflat[27]._super._super, arg0.kflat[28]._super._super, arg0.kflat[29]._super._super,
      arg0.kflat[30]._super._super, arg0.kflat[31]._super._super, arg0.kflat[32]._super._super,
      arg0.kflat[33]._super._super, arg0.kflat[34]._super._super, arg0.kflat[35]._super._super,
      arg0.kflat[36]._super._super, arg0.kflat[37]._super._super, arg0.kflat[38]._super._super,
      arg0.kflat[39]._super._super, arg0.kflat[40]._super._super, arg0.kflat[41]._super._super,
      arg0.kflat[42]._super._super, arg0.kflat[43]._super._super, arg0.kflat[44]._super._super,
      arg0.kflat[45]._super._super, arg0.kflat[46]._super._super, arg0.kflat[47]._super._super,
      arg0.kflat[48]._super._super, arg0.kflat[49]._super._super, arg0.kflat[50]._super._super,
      arg0.kflat[51]._super._super, arg0.kflat[52]._super._super, arg0.kflat[53]._super._super,
      arg0.kflat[54]._super._super, arg0.kflat[55]._super._super, arg0.kflat[56]._super._super,
      arg0.kflat[57]._super._super, arg0.kflat[58]._super._super, arg0.kflat[59]._super._super,
      arg0.kflat[60]._super._super, arg0.kflat[61]._super._super, arg0.kflat[62]._super._super,
      arg0.kflat[63]._super._super, arg0.kflat[64]._super._super, arg0.kflat[65]._super._super,
      arg0.kflat[66]._super._super, arg0.kflat[67]._super._super, arg0.kflat[68]._super._super,
      arg0.kflat[69]._super._super, arg0.kflat[70]._super._super, arg0.kflat[71]._super._super,
      arg0.kflat[72]._super._super, arg0.kflat[73]._super._super, arg0.kflat[74]._super._super,
      arg0.kflat[75]._super._super, arg0.kflat[76]._super._super, arg0.kflat[77]._super._super,
      arg0.kflat[78]._super._super, arg0.kflat[79]._super._super, arg0.kflat[80]._super._super,
      arg0.kflat[81]._super._super, arg0.kflat[82]._super._super, arg0.kflat[83]._super._super,
      arg0.kflat[84]._super._super, arg0.kflat[85]._super._super, arg0.kflat[86]._super._super,
      arg0.kflat[87]._super._super, arg0.kflat[88]._super._super, arg0.kflat[89]._super._super,
      arg0.kflat[90]._super._super, arg0.kflat[91]._super._super, arg0.kflat[92]._super._super,
      arg0.kflat[93]._super._super, arg0.kflat[94]._super._super, arg0.kflat[95]._super._super,
      arg0.kflat[96]._super._super, arg0.kflat[97]._super._super, arg0.kflat[98]._super._super,
      arg0.kflat[99]._super._super};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:253)
  Val16Array x683 = Val16Array{((x380 + x374) + x375),
                               ((x400 + x394) + x395),
                               ((x420 + x414) + x415),
                               ((x440 + x434) + x435),
                               ((x460 + x454) + x455),
                               ((x480 + x474) + x475),
                               ((x500 + x494) + x495),
                               ((x520 + x514) + x515),
                               ((x540 + x534) + x535),
                               ((x560 + x554) + x555),
                               ((x580 + x574) + x575),
                               ((x600 + x594) + x595),
                               ((x620 + x614) + x615),
                               ((x640 + x634) + x635),
                               ((x660 + x654) + x655),
                               ((x680 + x674) + x675)};
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:262)
  TopStateStruct x684 = exec_TopState(ctx, x681, x682, x683, LAYOUT_LOOKUP(layout1, _super));
  return x684;
}
__device__ WrapOneHotStruct back_WrapOneHot(ExecContext& ctx,
                                            Index distance0,
                                            BoundLayout<WrapOneHotLayout> layout1) {
  // WrapOneHot(zirgen/circuit/keccak2/top.zir:470)
  WrapOneHot_SuperStruct12Array x2 = map(
      Val12Array{Val(0),
                 Val(1),
                 Val(2),
                 Val(3),
                 Val(4),
                 Val(5),
                 Val(6),
                 Val(7),
                 Val(8),
                 Val(9),
                 Val(10),
                 Val(11)},
      LAYOUT_LOOKUP(layout1, _super),
      ([&](Val12Array::value_type x3, BoundLayout<NondetRegLayout12LayoutArray::value_type> x4) {
        NondetRegStruct x5 = back_Reg(ctx, distance0, x4);
        return WrapOneHot_SuperStruct{._super = x5};
      }));
  return WrapOneHotStruct{._super = x2};
}

} // namespace risc0::circuit::keccak::cuda
