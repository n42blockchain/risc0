#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"

namespace risc0::impl {

__device__ TopStateStruct exec_ReadCycle(ExecContext& ctx,
                                         TopStateStruct arg0,
                                         BoundLayout<ReadCycleLayout> layout1) {
  // ReadCycle(zirgen/circuit/keccak2/top.zir:333)
  ReadCycleLoadStruct100Array x2 = map(
      Val100Array{
          Val(0),  Val(1),  Val(2),  Val(3),  Val(4),  Val(5),  Val(6),  Val(7),  Val(8),  Val(9),
          Val(10), Val(11), Val(12), Val(13), Val(14), Val(15), Val(16), Val(17), Val(18), Val(19),
          Val(20), Val(21), Val(22), Val(23), Val(24), Val(25), Val(26), Val(27), Val(28), Val(29),
          Val(30), Val(31), Val(32), Val(33), Val(34), Val(35), Val(36), Val(37), Val(38), Val(39),
          Val(40), Val(41), Val(42), Val(43), Val(44), Val(45), Val(46), Val(47), Val(48), Val(49),
          Val(50), Val(51), Val(52), Val(53), Val(54), Val(55), Val(56), Val(57), Val(58), Val(59),
          Val(60), Val(61), Val(62), Val(63), Val(64), Val(65), Val(66), Val(67), Val(68), Val(69),
          Val(70), Val(71), Val(72), Val(73), Val(74), Val(75), Val(76), Val(77), Val(78), Val(79),
          Val(80), Val(81), Val(82), Val(83), Val(84), Val(85), Val(86), Val(87), Val(88), Val(89),
          Val(90), Val(91), Val(92), Val(93), Val(94), Val(95), Val(96), Val(97), Val(98), Val(99)},
      LAYOUT_LOOKUP(layout1, load),
      ([&](Val100Array::value_type x3, BoundLayout<NondetRegLayout100LayoutArray::value_type> x4) {
        // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
        Val x5 = INVOKE_EXTERN(ctx, getPreimage, x3);
        NondetRegStruct x6 = exec_NondetReg(ctx, x5, x4);
        return ReadCycleLoadStruct{._super = x6};
      }));
  // Log(<preamble>:22)
  // ReadCycle(zirgen/circuit/keccak2/top.zir:334)
  INVOKE_EXTERN(ctx, log, "ReadCycle", std::initializer_list<Val>{});
  // ReadCycle(zirgen/circuit/keccak2/top.zir:332)
  NondetRegStruct16Array x7 = arg0.sflat;
  Val16Array x8 = Val16Array{x7[0]._super,
                             x7[1]._super,
                             x7[2]._super,
                             x7[3]._super,
                             x7[4]._super,
                             x7[5]._super,
                             x7[6]._super,
                             x7[7]._super,
                             x7[8]._super,
                             x7[9]._super,
                             x7[10]._super,
                             x7[11]._super,
                             x7[12]._super,
                             x7[13]._super,
                             x7[14]._super,
                             x7[15]._super};
  // ReadCycle(zirgen/circuit/keccak2/top.zir:335)
  UnpackReg_800__16_Struct x9 = exec_SFlatToBits(ctx, x8, LAYOUT_LOOKUP(layout1, bits));
  UnpackReg_800__16__SuperStruct800Array x10 = x9._super;
  Val x11 = x10[0]._super._super;
  Val x12 = x10[1]._super._super;
  Val x13 = x10[2]._super._super;
  Val x14 = x10[3]._super._super;
  Val x15 = x10[4]._super._super;
  Val x16 = x10[5]._super._super;
  Val x17 = x10[6]._super._super;
  Val x18 = x10[7]._super._super;
  Val x19 = x10[8]._super._super;
  Val x20 = x10[9]._super._super;
  Val x21 = x10[10]._super._super;
  Val x22 = x10[11]._super._super;
  Val x23 = x10[12]._super._super;
  Val x24 = x10[13]._super._super;
  Val x25 = x10[14]._super._super;
  Val x26 = x10[15]._super._super;
  Val x27 = x10[16]._super._super;
  Val x28 = x10[17]._super._super;
  Val x29 = x10[18]._super._super;
  Val x30 = x10[19]._super._super;
  Val x31 = x10[20]._super._super;
  Val x32 = x10[21]._super._super;
  Val x33 = x10[22]._super._super;
  Val x34 = x10[23]._super._super;
  Val x35 = x10[24]._super._super;
  Val x36 = x10[25]._super._super;
  Val x37 = x10[26]._super._super;
  Val x38 = x10[27]._super._super;
  Val x39 = x10[28]._super._super;
  Val x40 = x10[29]._super._super;
  Val x41 = x10[30]._super._super;
  Val x42 = x10[31]._super._super;
  Val x43 = x10[32]._super._super;
  Val x44 = x10[33]._super._super;
  Val x45 = x10[34]._super._super;
  Val x46 = x10[35]._super._super;
  Val x47 = x10[36]._super._super;
  Val x48 = x10[37]._super._super;
  Val x49 = x10[38]._super._super;
  Val x50 = x10[39]._super._super;
  Val x51 = x10[40]._super._super;
  Val x52 = x10[41]._super._super;
  Val x53 = x10[42]._super._super;
  Val x54 = x10[43]._super._super;
  Val x55 = x10[44]._super._super;
  Val x56 = x10[45]._super._super;
  Val x57 = x10[46]._super._super;
  Val x58 = x10[47]._super._super;
  Val x59 = x10[48]._super._super;
  Val x60 = x10[49]._super._super;
  Val x61 = x10[50]._super._super;
  Val x62 = x10[51]._super._super;
  Val x63 = x10[52]._super._super;
  Val x64 = x10[53]._super._super;
  Val x65 = x10[54]._super._super;
  Val x66 = x10[55]._super._super;
  Val x67 = x10[56]._super._super;
  Val x68 = x10[57]._super._super;
  Val x69 = x10[58]._super._super;
  Val x70 = x10[59]._super._super;
  Val x71 = x10[60]._super._super;
  Val x72 = x10[61]._super._super;
  Val x73 = x10[62]._super._super;
  Val x74 = x10[63]._super._super;
  Val x75 = x10[64]._super._super;
  Val x76 = x10[65]._super._super;
  Val x77 = x10[66]._super._super;
  Val x78 = x10[67]._super._super;
  Val x79 = x10[68]._super._super;
  Val x80 = x10[69]._super._super;
  Val x81 = x10[70]._super._super;
  Val x82 = x10[71]._super._super;
  Val x83 = x10[72]._super._super;
  Val x84 = x10[73]._super._super;
  Val x85 = x10[74]._super._super;
  Val x86 = x10[75]._super._super;
  Val x87 = x10[76]._super._super;
  Val x88 = x10[77]._super._super;
  Val x89 = x10[78]._super._super;
  Val x90 = x10[79]._super._super;
  Val x91 = x10[80]._super._super;
  Val x92 = x10[81]._super._super;
  Val x93 = x10[82]._super._super;
  Val x94 = x10[83]._super._super;
  Val x95 = x10[84]._super._super;
  Val x96 = x10[85]._super._super;
  Val x97 = x10[86]._super._super;
  Val x98 = x10[87]._super._super;
  Val x99 = x10[88]._super._super;
  Val x100 = x10[89]._super._super;
  Val x101 = x10[90]._super._super;
  Val x102 = x10[91]._super._super;
  Val x103 = x10[92]._super._super;
  Val x104 = x10[93]._super._super;
  Val x105 = x10[94]._super._super;
  Val x106 = x10[95]._super._super;
  Val x107 = x10[96]._super._super;
  Val x108 = x10[97]._super._super;
  Val x109 = x10[98]._super._super;
  Val x110 = x10[99]._super._super;
  Val x111 = x10[100]._super._super;
  Val x112 = x10[101]._super._super;
  Val x113 = x10[102]._super._super;
  Val x114 = x10[103]._super._super;
  Val x115 = x10[104]._super._super;
  Val x116 = x10[105]._super._super;
  Val x117 = x10[106]._super._super;
  Val x118 = x10[107]._super._super;
  Val x119 = x10[108]._super._super;
  Val x120 = x10[109]._super._super;
  Val x121 = x10[110]._super._super;
  Val x122 = x10[111]._super._super;
  Val x123 = x10[112]._super._super;
  Val x124 = x10[113]._super._super;
  Val x125 = x10[114]._super._super;
  Val x126 = x10[115]._super._super;
  Val x127 = x10[116]._super._super;
  Val x128 = x10[117]._super._super;
  Val x129 = x10[118]._super._super;
  Val x130 = x10[119]._super._super;
  Val x131 = x10[120]._super._super;
  Val x132 = x10[121]._super._super;
  Val x133 = x10[122]._super._super;
  Val x134 = x10[123]._super._super;
  Val x135 = x10[124]._super._super;
  Val x136 = x10[125]._super._super;
  Val x137 = x10[126]._super._super;
  Val x138 = x10[127]._super._super;
  Val x139 = x10[128]._super._super;
  Val x140 = x10[129]._super._super;
  Val x141 = x10[130]._super._super;
  Val x142 = x10[131]._super._super;
  Val x143 = x10[132]._super._super;
  Val x144 = x10[133]._super._super;
  Val x145 = x10[134]._super._super;
  Val x146 = x10[135]._super._super;
  Val x147 = x10[136]._super._super;
  Val x148 = x10[137]._super._super;
  Val x149 = x10[138]._super._super;
  Val x150 = x10[139]._super._super;
  Val x151 = x10[140]._super._super;
  Val x152 = x10[141]._super._super;
  Val x153 = x10[142]._super._super;
  Val x154 = x10[143]._super._super;
  Val x155 = x10[144]._super._super;
  Val x156 = x10[145]._super._super;
  Val x157 = x10[146]._super._super;
  Val x158 = x10[147]._super._super;
  Val x159 = x10[148]._super._super;
  Val x160 = x10[149]._super._super;
  Val x161 = x10[150]._super._super;
  Val x162 = x10[151]._super._super;
  Val x163 = x10[152]._super._super;
  Val x164 = x10[153]._super._super;
  Val x165 = x10[154]._super._super;
  Val x166 = x10[155]._super._super;
  Val x167 = x10[156]._super._super;
  Val x168 = x10[157]._super._super;
  Val x169 = x10[158]._super._super;
  Val x170 = x10[159]._super._super;
  Val x171 = x10[160]._super._super;
  Val x172 = x10[161]._super._super;
  Val x173 = x10[162]._super._super;
  Val x174 = x10[163]._super._super;
  Val x175 = x10[164]._super._super;
  Val x176 = x10[165]._super._super;
  Val x177 = x10[166]._super._super;
  Val x178 = x10[167]._super._super;
  Val x179 = x10[168]._super._super;
  Val x180 = x10[169]._super._super;
  Val x181 = x10[170]._super._super;
  Val x182 = x10[171]._super._super;
  Val x183 = x10[172]._super._super;
  Val x184 = x10[173]._super._super;
  Val x185 = x10[174]._super._super;
  Val x186 = x10[175]._super._super;
  Val x187 = x10[176]._super._super;
  Val x188 = x10[177]._super._super;
  Val x189 = x10[178]._super._super;
  Val x190 = x10[179]._super._super;
  Val x191 = x10[180]._super._super;
  Val x192 = x10[181]._super._super;
  Val x193 = x10[182]._super._super;
  Val x194 = x10[183]._super._super;
  Val x195 = x10[184]._super._super;
  Val x196 = x10[185]._super._super;
  Val x197 = x10[186]._super._super;
  Val x198 = x10[187]._super._super;
  Val x199 = x10[188]._super._super;
  Val x200 = x10[189]._super._super;
  Val x201 = x10[190]._super._super;
  Val x202 = x10[191]._super._super;
  Val x203 = x10[192]._super._super;
  Val x204 = x10[193]._super._super;
  Val x205 = x10[194]._super._super;
  Val x206 = x10[195]._super._super;
  Val x207 = x10[196]._super._super;
  Val x208 = x10[197]._super._super;
  Val x209 = x10[198]._super._super;
  Val x210 = x10[199]._super._super;
  Val x211 = x10[200]._super._super;
  Val x212 = x10[201]._super._super;
  Val x213 = x10[202]._super._super;
  Val x214 = x10[203]._super._super;
  Val x215 = x10[204]._super._super;
  Val x216 = x10[205]._super._super;
  Val x217 = x10[206]._super._super;
  Val x218 = x10[207]._super._super;
  Val x219 = x10[208]._super._super;
  Val x220 = x10[209]._super._super;
  Val x221 = x10[210]._super._super;
  Val x222 = x10[211]._super._super;
  Val x223 = x10[212]._super._super;
  Val x224 = x10[213]._super._super;
  Val x225 = x10[214]._super._super;
  Val x226 = x10[215]._super._super;
  Val x227 = x10[216]._super._super;
  Val x228 = x10[217]._super._super;
  Val x229 = x10[218]._super._super;
  Val x230 = x10[219]._super._super;
  Val x231 = x10[220]._super._super;
  Val x232 = x10[221]._super._super;
  Val x233 = x10[222]._super._super;
  Val x234 = x10[223]._super._super;
  Val x235 = x10[224]._super._super;
  Val x236 = x10[225]._super._super;
  Val x237 = x10[226]._super._super;
  Val x238 = x10[227]._super._super;
  Val x239 = x10[228]._super._super;
  Val x240 = x10[229]._super._super;
  Val x241 = x10[230]._super._super;
  Val x242 = x10[231]._super._super;
  Val x243 = x10[232]._super._super;
  Val x244 = x10[233]._super._super;
  Val x245 = x10[234]._super._super;
  Val x246 = x10[235]._super._super;
  Val x247 = x10[236]._super._super;
  Val x248 = x10[237]._super._super;
  Val x249 = x10[238]._super._super;
  Val x250 = x10[239]._super._super;
  Val x251 = x10[240]._super._super;
  Val x252 = x10[241]._super._super;
  Val x253 = x10[242]._super._super;
  Val x254 = x10[243]._super._super;
  Val x255 = x10[244]._super._super;
  Val x256 = x10[245]._super._super;
  Val x257 = x10[246]._super._super;
  Val x258 = x10[247]._super._super;
  Val x259 = x10[248]._super._super;
  Val x260 = x10[249]._super._super;
  Val x261 = x10[250]._super._super;
  Val x262 = x10[251]._super._super;
  Val x263 = x10[252]._super._super;
  Val x264 = x10[253]._super._super;
  Val x265 = x10[254]._super._super;
  Val x266 = x10[255]._super._super;
  Val x267 = x10[256]._super._super;
  Val x268 = x10[257]._super._super;
  Val x269 = x10[258]._super._super;
  Val x270 = x10[259]._super._super;
  Val x271 = x10[260]._super._super;
  Val x272 = x10[261]._super._super;
  Val x273 = x10[262]._super._super;
  Val x274 = x10[263]._super._super;
  Val x275 = x10[264]._super._super;
  Val x276 = x10[265]._super._super;
  Val x277 = x10[266]._super._super;
  Val x278 = x10[267]._super._super;
  Val x279 = x10[268]._super._super;
  Val x280 = x10[269]._super._super;
  Val x281 = x10[270]._super._super;
  Val x282 = x10[271]._super._super;
  Val x283 = x10[272]._super._super;
  Val x284 = x10[273]._super._super;
  Val x285 = x10[274]._super._super;
  Val x286 = x10[275]._super._super;
  Val x287 = x10[276]._super._super;
  Val x288 = x10[277]._super._super;
  Val x289 = x10[278]._super._super;
  Val x290 = x10[279]._super._super;
  Val x291 = x10[280]._super._super;
  Val x292 = x10[281]._super._super;
  Val x293 = x10[282]._super._super;
  Val x294 = x10[283]._super._super;
  Val x295 = x10[284]._super._super;
  Val x296 = x10[285]._super._super;
  Val x297 = x10[286]._super._super;
  Val x298 = x10[287]._super._super;
  Val x299 = x10[288]._super._super;
  Val x300 = x10[289]._super._super;
  Val x301 = x10[290]._super._super;
  Val x302 = x10[291]._super._super;
  Val x303 = x10[292]._super._super;
  Val x304 = x10[293]._super._super;
  Val x305 = x10[294]._super._super;
  Val x306 = x10[295]._super._super;
  Val x307 = x10[296]._super._super;
  Val x308 = x10[297]._super._super;
  Val x309 = x10[298]._super._super;
  Val x310 = x10[299]._super._super;
  Val x311 = x10[300]._super._super;
  Val x312 = x10[301]._super._super;
  Val x313 = x10[302]._super._super;
  Val x314 = x10[303]._super._super;
  Val x315 = x10[304]._super._super;
  Val x316 = x10[305]._super._super;
  Val x317 = x10[306]._super._super;
  Val x318 = x10[307]._super._super;
  Val x319 = x10[308]._super._super;
  Val x320 = x10[309]._super._super;
  Val x321 = x10[310]._super._super;
  Val x322 = x10[311]._super._super;
  Val x323 = x10[312]._super._super;
  Val x324 = x10[313]._super._super;
  Val x325 = x10[314]._super._super;
  Val x326 = x10[315]._super._super;
  Val x327 = x10[316]._super._super;
  Val x328 = x10[317]._super._super;
  Val x329 = x10[318]._super._super;
  Val x330 = x10[319]._super._super;
  Val x331 = x10[320]._super._super;
  Val x332 = x10[321]._super._super;
  Val x333 = x10[322]._super._super;
  Val x334 = x10[323]._super._super;
  Val x335 = x10[324]._super._super;
  Val x336 = x10[325]._super._super;
  Val x337 = x10[326]._super._super;
  Val x338 = x10[327]._super._super;
  Val x339 = x10[328]._super._super;
  Val x340 = x10[329]._super._super;
  Val x341 = x10[330]._super._super;
  Val x342 = x10[331]._super._super;
  Val x343 = x10[332]._super._super;
  Val x344 = x10[333]._super._super;
  Val x345 = x10[334]._super._super;
  Val x346 = x10[335]._super._super;
  Val x347 = x10[336]._super._super;
  Val x348 = x10[337]._super._super;
  Val x349 = x10[338]._super._super;
  Val x350 = x10[339]._super._super;
  Val x351 = x10[340]._super._super;
  Val x352 = x10[341]._super._super;
  Val x353 = x10[342]._super._super;
  Val x354 = x10[343]._super._super;
  Val x355 = x10[344]._super._super;
  Val x356 = x10[345]._super._super;
  Val x357 = x10[346]._super._super;
  Val x358 = x10[347]._super._super;
  Val x359 = x10[348]._super._super;
  Val x360 = x10[349]._super._super;
  Val x361 = x10[350]._super._super;
  Val x362 = x10[351]._super._super;
  Val x363 = x10[352]._super._super;
  Val x364 = x10[353]._super._super;
  Val x365 = x10[354]._super._super;
  Val x366 = x10[355]._super._super;
  Val x367 = x10[356]._super._super;
  Val x368 = x10[357]._super._super;
  Val x369 = x10[358]._super._super;
  Val x370 = x10[359]._super._super;
  Val x371 = x10[360]._super._super;
  Val x372 = x10[361]._super._super;
  Val x373 = x10[362]._super._super;
  Val x374 = x10[363]._super._super;
  Val x375 = x10[364]._super._super;
  Val x376 = x10[365]._super._super;
  Val x377 = x10[366]._super._super;
  Val x378 = x10[367]._super._super;
  Val x379 = x10[368]._super._super;
  Val x380 = x10[369]._super._super;
  Val x381 = x10[370]._super._super;
  Val x382 = x10[371]._super._super;
  Val x383 = x10[372]._super._super;
  Val x384 = x10[373]._super._super;
  Val x385 = x10[374]._super._super;
  Val x386 = x10[375]._super._super;
  Val x387 = x10[376]._super._super;
  Val x388 = x10[377]._super._super;
  Val x389 = x10[378]._super._super;
  Val x390 = x10[379]._super._super;
  Val x391 = x10[380]._super._super;
  Val x392 = x10[381]._super._super;
  Val x393 = x10[382]._super._super;
  Val x394 = x10[383]._super._super;
  Val x395 = x10[384]._super._super;
  Val x396 = x10[385]._super._super;
  Val x397 = x10[386]._super._super;
  Val x398 = x10[387]._super._super;
  Val x399 = x10[388]._super._super;
  Val x400 = x10[389]._super._super;
  Val x401 = x10[390]._super._super;
  Val x402 = x10[391]._super._super;
  Val x403 = x10[392]._super._super;
  Val x404 = x10[393]._super._super;
  Val x405 = x10[394]._super._super;
  Val x406 = x10[395]._super._super;
  Val x407 = x10[396]._super._super;
  Val x408 = x10[397]._super._super;
  Val x409 = x10[398]._super._super;
  Val x410 = x10[399]._super._super;
  Val x411 = x10[400]._super._super;
  Val x412 = x10[401]._super._super;
  Val x413 = x10[402]._super._super;
  Val x414 = x10[403]._super._super;
  Val x415 = x10[404]._super._super;
  Val x416 = x10[405]._super._super;
  Val x417 = x10[406]._super._super;
  Val x418 = x10[407]._super._super;
  Val x419 = x10[408]._super._super;
  Val x420 = x10[409]._super._super;
  Val x421 = x10[410]._super._super;
  Val x422 = x10[411]._super._super;
  Val x423 = x10[412]._super._super;
  Val x424 = x10[413]._super._super;
  Val x425 = x10[414]._super._super;
  Val x426 = x10[415]._super._super;
  Val x427 = x10[416]._super._super;
  Val x428 = x10[417]._super._super;
  Val x429 = x10[418]._super._super;
  Val x430 = x10[419]._super._super;
  Val x431 = x10[420]._super._super;
  Val x432 = x10[421]._super._super;
  Val x433 = x10[422]._super._super;
  Val x434 = x10[423]._super._super;
  Val x435 = x10[424]._super._super;
  Val x436 = x10[425]._super._super;
  Val x437 = x10[426]._super._super;
  Val x438 = x10[427]._super._super;
  Val x439 = x10[428]._super._super;
  Val x440 = x10[429]._super._super;
  Val x441 = x10[430]._super._super;
  Val x442 = x10[431]._super._super;
  Val x443 = x10[432]._super._super;
  Val x444 = x10[433]._super._super;
  Val x445 = x10[434]._super._super;
  Val x446 = x10[435]._super._super;
  Val x447 = x10[436]._super._super;
  Val x448 = x10[437]._super._super;
  Val x449 = x10[438]._super._super;
  Val x450 = x10[439]._super._super;
  Val x451 = x10[440]._super._super;
  Val x452 = x10[441]._super._super;
  Val x453 = x10[442]._super._super;
  Val x454 = x10[443]._super._super;
  Val x455 = x10[444]._super._super;
  Val x456 = x10[445]._super._super;
  Val x457 = x10[446]._super._super;
  Val x458 = x10[447]._super._super;
  Val x459 = x10[448]._super._super;
  Val x460 = x10[449]._super._super;
  Val x461 = x10[450]._super._super;
  Val x462 = x10[451]._super._super;
  Val x463 = x10[452]._super._super;
  Val x464 = x10[453]._super._super;
  Val x465 = x10[454]._super._super;
  Val x466 = x10[455]._super._super;
  Val x467 = x10[456]._super._super;
  Val x468 = x10[457]._super._super;
  Val x469 = x10[458]._super._super;
  Val x470 = x10[459]._super._super;
  Val x471 = x10[460]._super._super;
  Val x472 = x10[461]._super._super;
  Val x473 = x10[462]._super._super;
  Val x474 = x10[463]._super._super;
  Val x475 = x10[464]._super._super;
  Val x476 = x10[465]._super._super;
  Val x477 = x10[466]._super._super;
  Val x478 = x10[467]._super._super;
  Val x479 = x10[468]._super._super;
  Val x480 = x10[469]._super._super;
  Val x481 = x10[470]._super._super;
  Val x482 = x10[471]._super._super;
  Val x483 = x10[472]._super._super;
  Val x484 = x10[473]._super._super;
  Val x485 = x10[474]._super._super;
  Val x486 = x10[475]._super._super;
  Val x487 = x10[476]._super._super;
  Val x488 = x10[477]._super._super;
  Val x489 = x10[478]._super._super;
  Val x490 = x10[479]._super._super;
  Val x491 = x10[480]._super._super;
  Val x492 = x10[481]._super._super;
  Val x493 = x10[482]._super._super;
  Val x494 = x10[483]._super._super;
  Val x495 = x10[484]._super._super;
  Val x496 = x10[485]._super._super;
  Val x497 = x10[486]._super._super;
  Val x498 = x10[487]._super._super;
  Val x499 = x10[488]._super._super;
  Val x500 = x10[489]._super._super;
  Val x501 = x10[490]._super._super;
  Val x502 = x10[491]._super._super;
  Val x503 = x10[492]._super._super;
  Val x504 = x10[493]._super._super;
  Val x505 = x10[494]._super._super;
  Val x506 = x10[495]._super._super;
  Val x507 = x10[496]._super._super;
  Val x508 = x10[497]._super._super;
  Val x509 = x10[498]._super._super;
  Val x510 = x10[499]._super._super;
  Val x511 = x10[500]._super._super;
  Val x512 = x10[501]._super._super;
  Val x513 = x10[502]._super._super;
  Val x514 = x10[503]._super._super;
  Val x515 = x10[504]._super._super;
  Val x516 = x10[505]._super._super;
  Val x517 = x10[506]._super._super;
  Val x518 = x10[507]._super._super;
  Val x519 = x10[508]._super._super;
  Val x520 = x10[509]._super._super;
  Val x521 = x10[510]._super._super;
  Val x522 = x10[511]._super._super;
  Val x523 = x10[512]._super._super;
  Val x524 = x10[513]._super._super;
  Val x525 = x10[514]._super._super;
  Val x526 = x10[515]._super._super;
  Val x527 = x10[516]._super._super;
  Val x528 = x10[517]._super._super;
  Val x529 = x10[518]._super._super;
  Val x530 = x10[519]._super._super;
  Val x531 = x10[520]._super._super;
  Val x532 = x10[521]._super._super;
  Val x533 = x10[522]._super._super;
  Val x534 = x10[523]._super._super;
  Val x535 = x10[524]._super._super;
  Val x536 = x10[525]._super._super;
  Val x537 = x10[526]._super._super;
  Val x538 = x10[527]._super._super;
  Val x539 = x10[528]._super._super;
  Val x540 = x10[529]._super._super;
  Val x541 = x10[530]._super._super;
  Val x542 = x10[531]._super._super;
  Val x543 = x10[532]._super._super;
  Val x544 = x10[533]._super._super;
  Val x545 = x10[534]._super._super;
  Val x546 = x10[535]._super._super;
  Val x547 = x10[536]._super._super;
  Val x548 = x10[537]._super._super;
  Val x549 = x10[538]._super._super;
  Val x550 = x10[539]._super._super;
  Val x551 = x10[540]._super._super;
  Val x552 = x10[541]._super._super;
  Val x553 = x10[542]._super._super;
  Val x554 = x10[543]._super._super;
  Val x555 = x10[544]._super._super;
  Val x556 = x10[545]._super._super;
  Val x557 = x10[546]._super._super;
  Val x558 = x10[547]._super._super;
  Val x559 = x10[548]._super._super;
  Val x560 = x10[549]._super._super;
  Val x561 = x10[550]._super._super;
  Val x562 = x10[551]._super._super;
  Val x563 = x10[552]._super._super;
  Val x564 = x10[553]._super._super;
  Val x565 = x10[554]._super._super;
  Val x566 = x10[555]._super._super;
  Val x567 = x10[556]._super._super;
  Val x568 = x10[557]._super._super;
  Val x569 = x10[558]._super._super;
  Val x570 = x10[559]._super._super;
  Val x571 = x10[560]._super._super;
  Val x572 = x10[561]._super._super;
  Val x573 = x10[562]._super._super;
  Val x574 = x10[563]._super._super;
  Val x575 = x10[564]._super._super;
  Val x576 = x10[565]._super._super;
  Val x577 = x10[566]._super._super;
  Val x578 = x10[567]._super._super;
  Val x579 = x10[568]._super._super;
  Val x580 = x10[569]._super._super;
  Val x581 = x10[570]._super._super;
  Val x582 = x10[571]._super._super;
  Val x583 = x10[572]._super._super;
  Val x584 = x10[573]._super._super;
  Val x585 = x10[574]._super._super;
  Val x586 = x10[575]._super._super;
  Val x587 = x10[576]._super._super;
  Val x588 = x10[577]._super._super;
  Val x589 = x10[578]._super._super;
  Val x590 = x10[579]._super._super;
  Val x591 = x10[580]._super._super;
  Val x592 = x10[581]._super._super;
  Val x593 = x10[582]._super._super;
  Val x594 = x10[583]._super._super;
  Val x595 = x10[584]._super._super;
  Val x596 = x10[585]._super._super;
  Val x597 = x10[586]._super._super;
  Val x598 = x10[587]._super._super;
  Val x599 = x10[588]._super._super;
  Val x600 = x10[589]._super._super;
  Val x601 = x10[590]._super._super;
  Val x602 = x10[591]._super._super;
  Val x603 = x10[592]._super._super;
  Val x604 = x10[593]._super._super;
  Val x605 = x10[594]._super._super;
  Val x606 = x10[595]._super._super;
  Val x607 = x10[596]._super._super;
  Val x608 = x10[597]._super._super;
  Val x609 = x10[598]._super._super;
  Val x610 = x10[599]._super._super;
  Val x611 = x10[600]._super._super;
  Val x612 = x10[601]._super._super;
  Val x613 = x10[602]._super._super;
  Val x614 = x10[603]._super._super;
  Val x615 = x10[604]._super._super;
  Val x616 = x10[605]._super._super;
  Val x617 = x10[606]._super._super;
  Val x618 = x10[607]._super._super;
  Val x619 = x10[608]._super._super;
  Val x620 = x10[609]._super._super;
  Val x621 = x10[610]._super._super;
  Val x622 = x10[611]._super._super;
  Val x623 = x10[612]._super._super;
  Val x624 = x10[613]._super._super;
  Val x625 = x10[614]._super._super;
  Val x626 = x10[615]._super._super;
  Val x627 = x10[616]._super._super;
  Val x628 = x10[617]._super._super;
  Val x629 = x10[618]._super._super;
  Val x630 = x10[619]._super._super;
  Val x631 = x10[620]._super._super;
  Val x632 = x10[621]._super._super;
  Val x633 = x10[622]._super._super;
  Val x634 = x10[623]._super._super;
  Val x635 = x10[624]._super._super;
  Val x636 = x10[625]._super._super;
  Val x637 = x10[626]._super._super;
  Val x638 = x10[627]._super._super;
  Val x639 = x10[628]._super._super;
  Val x640 = x10[629]._super._super;
  Val x641 = x10[630]._super._super;
  Val x642 = x10[631]._super._super;
  Val x643 = x10[632]._super._super;
  Val x644 = x10[633]._super._super;
  Val x645 = x10[634]._super._super;
  Val x646 = x10[635]._super._super;
  Val x647 = x10[636]._super._super;
  Val x648 = x10[637]._super._super;
  Val x649 = x10[638]._super._super;
  Val x650 = x10[639]._super._super;
  Val x651 = x10[640]._super._super;
  Val x652 = x10[641]._super._super;
  Val x653 = x10[642]._super._super;
  Val x654 = x10[643]._super._super;
  Val x655 = x10[644]._super._super;
  Val x656 = x10[645]._super._super;
  Val x657 = x10[646]._super._super;
  Val x658 = x10[647]._super._super;
  Val x659 = x10[648]._super._super;
  Val x660 = x10[649]._super._super;
  Val x661 = x10[650]._super._super;
  Val x662 = x10[651]._super._super;
  Val x663 = x10[652]._super._super;
  Val x664 = x10[653]._super._super;
  Val x665 = x10[654]._super._super;
  Val x666 = x10[655]._super._super;
  Val x667 = x10[656]._super._super;
  Val x668 = x10[657]._super._super;
  Val x669 = x10[658]._super._super;
  Val x670 = x10[659]._super._super;
  Val x671 = x10[660]._super._super;
  Val x672 = x10[661]._super._super;
  Val x673 = x10[662]._super._super;
  Val x674 = x10[663]._super._super;
  Val x675 = x10[664]._super._super;
  Val x676 = x10[665]._super._super;
  Val x677 = x10[666]._super._super;
  Val x678 = x10[667]._super._super;
  Val x679 = x10[668]._super._super;
  Val x680 = x10[669]._super._super;
  Val x681 = x10[670]._super._super;
  Val x682 = x10[671]._super._super;
  Val x683 = x10[672]._super._super;
  Val x684 = x10[673]._super._super;
  Val x685 = x10[674]._super._super;
  Val x686 = x10[675]._super._super;
  Val x687 = x10[676]._super._super;
  Val x688 = x10[677]._super._super;
  Val x689 = x10[678]._super._super;
  Val x690 = x10[679]._super._super;
  Val x691 = x10[680]._super._super;
  Val x692 = x10[681]._super._super;
  Val x693 = x10[682]._super._super;
  Val x694 = x10[683]._super._super;
  Val x695 = x10[684]._super._super;
  Val x696 = x10[685]._super._super;
  Val x697 = x10[686]._super._super;
  Val x698 = x10[687]._super._super;
  Val x699 = x10[688]._super._super;
  Val x700 = x10[689]._super._super;
  Val x701 = x10[690]._super._super;
  Val x702 = x10[691]._super._super;
  Val x703 = x10[692]._super._super;
  Val x704 = x10[693]._super._super;
  Val x705 = x10[694]._super._super;
  Val x706 = x10[695]._super._super;
  Val x707 = x10[696]._super._super;
  Val x708 = x10[697]._super._super;
  Val x709 = x10[698]._super._super;
  Val x710 = x10[699]._super._super;
  Val x711 = x10[700]._super._super;
  Val x712 = x10[701]._super._super;
  Val x713 = x10[702]._super._super;
  Val x714 = x10[703]._super._super;
  Val x715 = x10[704]._super._super;
  Val x716 = x10[705]._super._super;
  Val x717 = x10[706]._super._super;
  Val x718 = x10[707]._super._super;
  Val x719 = x10[708]._super._super;
  Val x720 = x10[709]._super._super;
  Val x721 = x10[710]._super._super;
  Val x722 = x10[711]._super._super;
  Val x723 = x10[712]._super._super;
  Val x724 = x10[713]._super._super;
  Val x725 = x10[714]._super._super;
  Val x726 = x10[715]._super._super;
  Val x727 = x10[716]._super._super;
  Val x728 = x10[717]._super._super;
  Val x729 = x10[718]._super._super;
  Val x730 = x10[719]._super._super;
  Val x731 = x10[720]._super._super;
  Val x732 = x10[721]._super._super;
  Val x733 = x10[722]._super._super;
  Val x734 = x10[723]._super._super;
  Val x735 = x10[724]._super._super;
  Val x736 = x10[725]._super._super;
  Val x737 = x10[726]._super._super;
  Val x738 = x10[727]._super._super;
  Val x739 = x10[728]._super._super;
  Val x740 = x10[729]._super._super;
  Val x741 = x10[730]._super._super;
  Val x742 = x10[731]._super._super;
  Val x743 = x10[732]._super._super;
  Val x744 = x10[733]._super._super;
  Val x745 = x10[734]._super._super;
  Val x746 = x10[735]._super._super;
  Val x747 = x10[736]._super._super;
  Val x748 = x10[737]._super._super;
  Val x749 = x10[738]._super._super;
  Val x750 = x10[739]._super._super;
  Val x751 = x10[740]._super._super;
  Val x752 = x10[741]._super._super;
  Val x753 = x10[742]._super._super;
  Val x754 = x10[743]._super._super;
  Val x755 = x10[744]._super._super;
  Val x756 = x10[745]._super._super;
  Val x757 = x10[746]._super._super;
  Val x758 = x10[747]._super._super;
  Val x759 = x10[748]._super._super;
  Val x760 = x10[749]._super._super;
  Val x761 = x10[750]._super._super;
  Val x762 = x10[751]._super._super;
  Val x763 = x10[752]._super._super;
  Val x764 = x10[753]._super._super;
  Val x765 = x10[754]._super._super;
  Val x766 = x10[755]._super._super;
  Val x767 = x10[756]._super._super;
  Val x768 = x10[757]._super._super;
  Val x769 = x10[758]._super._super;
  Val x770 = x10[759]._super._super;
  Val x771 = x10[760]._super._super;
  Val x772 = x10[761]._super._super;
  Val x773 = x10[762]._super._super;
  Val x774 = x10[763]._super._super;
  Val x775 = x10[764]._super._super;
  Val x776 = x10[765]._super._super;
  Val x777 = x10[766]._super._super;
  Val x778 = x10[767]._super._super;
  Val x779 = x10[768]._super._super;
  Val x780 = x10[769]._super._super;
  Val x781 = x10[770]._super._super;
  Val x782 = x10[771]._super._super;
  Val x783 = x10[772]._super._super;
  Val x784 = x10[773]._super._super;
  Val x785 = x10[774]._super._super;
  Val x786 = x10[775]._super._super;
  Val x787 = x10[776]._super._super;
  Val x788 = x10[777]._super._super;
  Val x789 = x10[778]._super._super;
  Val x790 = x10[779]._super._super;
  Val x791 = x10[780]._super._super;
  Val x792 = x10[781]._super._super;
  Val x793 = x10[782]._super._super;
  Val x794 = x10[783]._super._super;
  Val x795 = x10[784]._super._super;
  Val x796 = x10[785]._super._super;
  Val x797 = x10[786]._super._super;
  Val x798 = x10[787]._super._super;
  Val x799 = x10[788]._super._super;
  Val x800 = x10[789]._super._super;
  Val x801 = x10[790]._super._super;
  Val x802 = x10[791]._super._super;
  Val x803 = x10[792]._super._super;
  Val x804 = x10[793]._super._super;
  Val x805 = x10[794]._super._super;
  Val x806 = x10[795]._super._super;
  Val x807 = x10[796]._super._super;
  Val x808 = x10[797]._super._super;
  Val x809 = x10[798]._super._super;
  Val x810 = x10[799]._super._super;
  // ReadCycle(zirgen/circuit/keccak2/top.zir:333)
  Val x811 = x2[0]._super._super;
  Val x812 = x2[1]._super._super;
  Val x813 = x2[2]._super._super;
  Val x814 = x2[3]._super._super;
  Val x815 = x2[4]._super._super;
  Val x816 = x2[5]._super._super;
  Val x817 = x2[6]._super._super;
  Val x818 = x2[7]._super._super;
  Val x819 = x2[8]._super._super;
  Val x820 = x2[9]._super._super;
  Val x821 = x2[10]._super._super;
  Val x822 = x2[11]._super._super;
  Val x823 = x2[12]._super._super;
  Val x824 = x2[13]._super._super;
  Val x825 = x2[14]._super._super;
  Val x826 = x2[15]._super._super;
  Val x827 = x2[16]._super._super;
  Val x828 = x2[17]._super._super;
  Val x829 = x2[18]._super._super;
  Val x830 = x2[19]._super._super;
  Val x831 = x2[20]._super._super;
  Val x832 = x2[21]._super._super;
  Val x833 = x2[22]._super._super;
  Val x834 = x2[23]._super._super;
  Val x835 = x2[24]._super._super;
  Val x836 = x2[25]._super._super;
  Val x837 = x2[26]._super._super;
  Val x838 = x2[27]._super._super;
  Val x839 = x2[28]._super._super;
  Val x840 = x2[29]._super._super;
  Val x841 = x2[30]._super._super;
  Val x842 = x2[31]._super._super;
  Val x843 = x2[32]._super._super;
  Val x844 = x2[33]._super._super;
  Val x845 = x2[34]._super._super;
  Val x846 = x2[35]._super._super;
  Val x847 = x2[36]._super._super;
  Val x848 = x2[37]._super._super;
  Val x849 = x2[38]._super._super;
  Val x850 = x2[39]._super._super;
  Val x851 = x2[40]._super._super;
  Val x852 = x2[41]._super._super;
  Val x853 = x2[42]._super._super;
  Val x854 = x2[43]._super._super;
  Val x855 = x2[44]._super._super;
  Val x856 = x2[45]._super._super;
  Val x857 = x2[46]._super._super;
  Val x858 = x2[47]._super._super;
  Val x859 = x2[48]._super._super;
  Val x860 = x2[49]._super._super;
  Val x861 = x2[50]._super._super;
  Val x862 = x2[51]._super._super;
  Val x863 = x2[52]._super._super;
  Val x864 = x2[53]._super._super;
  Val x865 = x2[54]._super._super;
  Val x866 = x2[55]._super._super;
  Val x867 = x2[56]._super._super;
  Val x868 = x2[57]._super._super;
  Val x869 = x2[58]._super._super;
  Val x870 = x2[59]._super._super;
  Val x871 = x2[60]._super._super;
  Val x872 = x2[61]._super._super;
  Val x873 = x2[62]._super._super;
  Val x874 = x2[63]._super._super;
  Val x875 = x2[64]._super._super;
  Val x876 = x2[65]._super._super;
  Val x877 = x2[66]._super._super;
  Val x878 = x2[67]._super._super;
  Val x879 = x2[68]._super._super;
  Val x880 = x2[69]._super._super;
  Val x881 = x2[70]._super._super;
  Val x882 = x2[71]._super._super;
  Val x883 = x2[72]._super._super;
  Val x884 = x2[73]._super._super;
  Val x885 = x2[74]._super._super;
  Val x886 = x2[75]._super._super;
  Val x887 = x2[76]._super._super;
  Val x888 = x2[77]._super._super;
  Val x889 = x2[78]._super._super;
  Val x890 = x2[79]._super._super;
  Val x891 = x2[80]._super._super;
  Val x892 = x2[81]._super._super;
  Val x893 = x2[82]._super._super;
  Val x894 = x2[83]._super._super;
  Val x895 = x2[84]._super._super;
  Val x896 = x2[85]._super._super;
  Val x897 = x2[86]._super._super;
  Val x898 = x2[87]._super._super;
  Val x899 = x2[88]._super._super;
  Val x900 = x2[89]._super._super;
  Val x901 = x2[90]._super._super;
  Val x902 = x2[91]._super._super;
  Val x903 = x2[92]._super._super;
  Val x904 = x2[93]._super._super;
  Val x905 = x2[94]._super._super;
  Val x906 = x2[95]._super._super;
  Val x907 = x2[96]._super._super;
  Val x908 = x2[97]._super._super;
  Val x909 = x2[98]._super._super;
  Val x910 = x2[99]._super._super;
  // ReadCycle(zirgen/circuit/keccak2/top.zir:336)
  TopStateStruct x911 = exec_TopState(
      ctx,
      Val800Array{
          x11,  x12,  x13,  x14,  x15,  x16,  x17,  x18,  x19,  x20,  x21,  x22,  x23,  x24,  x25,
          x26,  x27,  x28,  x29,  x30,  x31,  x32,  x33,  x34,  x35,  x36,  x37,  x38,  x39,  x40,
          x41,  x42,  x43,  x44,  x45,  x46,  x47,  x48,  x49,  x50,  x51,  x52,  x53,  x54,  x55,
          x56,  x57,  x58,  x59,  x60,  x61,  x62,  x63,  x64,  x65,  x66,  x67,  x68,  x69,  x70,
          x71,  x72,  x73,  x74,  x75,  x76,  x77,  x78,  x79,  x80,  x81,  x82,  x83,  x84,  x85,
          x86,  x87,  x88,  x89,  x90,  x91,  x92,  x93,  x94,  x95,  x96,  x97,  x98,  x99,  x100,
          x101, x102, x103, x104, x105, x106, x107, x108, x109, x110, x111, x112, x113, x114, x115,
          x116, x117, x118, x119, x120, x121, x122, x123, x124, x125, x126, x127, x128, x129, x130,
          x131, x132, x133, x134, x135, x136, x137, x138, x139, x140, x141, x142, x143, x144, x145,
          x146, x147, x148, x149, x150, x151, x152, x153, x154, x155, x156, x157, x158, x159, x160,
          x161, x162, x163, x164, x165, x166, x167, x168, x169, x170, x171, x172, x173, x174, x175,
          x176, x177, x178, x179, x180, x181, x182, x183, x184, x185, x186, x187, x188, x189, x190,
          x191, x192, x193, x194, x195, x196, x197, x198, x199, x200, x201, x202, x203, x204, x205,
          x206, x207, x208, x209, x210, x211, x212, x213, x214, x215, x216, x217, x218, x219, x220,
          x221, x222, x223, x224, x225, x226, x227, x228, x229, x230, x231, x232, x233, x234, x235,
          x236, x237, x238, x239, x240, x241, x242, x243, x244, x245, x246, x247, x248, x249, x250,
          x251, x252, x253, x254, x255, x256, x257, x258, x259, x260, x261, x262, x263, x264, x265,
          x266, x267, x268, x269, x270, x271, x272, x273, x274, x275, x276, x277, x278, x279, x280,
          x281, x282, x283, x284, x285, x286, x287, x288, x289, x290, x291, x292, x293, x294, x295,
          x296, x297, x298, x299, x300, x301, x302, x303, x304, x305, x306, x307, x308, x309, x310,
          x311, x312, x313, x314, x315, x316, x317, x318, x319, x320, x321, x322, x323, x324, x325,
          x326, x327, x328, x329, x330, x331, x332, x333, x334, x335, x336, x337, x338, x339, x340,
          x341, x342, x343, x344, x345, x346, x347, x348, x349, x350, x351, x352, x353, x354, x355,
          x356, x357, x358, x359, x360, x361, x362, x363, x364, x365, x366, x367, x368, x369, x370,
          x371, x372, x373, x374, x375, x376, x377, x378, x379, x380, x381, x382, x383, x384, x385,
          x386, x387, x388, x389, x390, x391, x392, x393, x394, x395, x396, x397, x398, x399, x400,
          x401, x402, x403, x404, x405, x406, x407, x408, x409, x410, x411, x412, x413, x414, x415,
          x416, x417, x418, x419, x420, x421, x422, x423, x424, x425, x426, x427, x428, x429, x430,
          x431, x432, x433, x434, x435, x436, x437, x438, x439, x440, x441, x442, x443, x444, x445,
          x446, x447, x448, x449, x450, x451, x452, x453, x454, x455, x456, x457, x458, x459, x460,
          x461, x462, x463, x464, x465, x466, x467, x468, x469, x470, x471, x472, x473, x474, x475,
          x476, x477, x478, x479, x480, x481, x482, x483, x484, x485, x486, x487, x488, x489, x490,
          x491, x492, x493, x494, x495, x496, x497, x498, x499, x500, x501, x502, x503, x504, x505,
          x506, x507, x508, x509, x510, x511, x512, x513, x514, x515, x516, x517, x518, x519, x520,
          x521, x522, x523, x524, x525, x526, x527, x528, x529, x530, x531, x532, x533, x534, x535,
          x536, x537, x538, x539, x540, x541, x542, x543, x544, x545, x546, x547, x548, x549, x550,
          x551, x552, x553, x554, x555, x556, x557, x558, x559, x560, x561, x562, x563, x564, x565,
          x566, x567, x568, x569, x570, x571, x572, x573, x574, x575, x576, x577, x578, x579, x580,
          x581, x582, x583, x584, x585, x586, x587, x588, x589, x590, x591, x592, x593, x594, x595,
          x596, x597, x598, x599, x600, x601, x602, x603, x604, x605, x606, x607, x608, x609, x610,
          x611, x612, x613, x614, x615, x616, x617, x618, x619, x620, x621, x622, x623, x624, x625,
          x626, x627, x628, x629, x630, x631, x632, x633, x634, x635, x636, x637, x638, x639, x640,
          x641, x642, x643, x644, x645, x646, x647, x648, x649, x650, x651, x652, x653, x654, x655,
          x656, x657, x658, x659, x660, x661, x662, x663, x664, x665, x666, x667, x668, x669, x670,
          x671, x672, x673, x674, x675, x676, x677, x678, x679, x680, x681, x682, x683, x684, x685,
          x686, x687, x688, x689, x690, x691, x692, x693, x694, x695, x696, x697, x698, x699, x700,
          x701, x702, x703, x704, x705, x706, x707, x708, x709, x710, x711, x712, x713, x714, x715,
          x716, x717, x718, x719, x720, x721, x722, x723, x724, x725, x726, x727, x728, x729, x730,
          x731, x732, x733, x734, x735, x736, x737, x738, x739, x740, x741, x742, x743, x744, x745,
          x746, x747, x748, x749, x750, x751, x752, x753, x754, x755, x756, x757, x758, x759, x760,
          x761, x762, x763, x764, x765, x766, x767, x768, x769, x770, x771, x772, x773, x774, x775,
          x776, x777, x778, x779, x780, x781, x782, x783, x784, x785, x786, x787, x788, x789, x790,
          x791, x792, x793, x794, x795, x796, x797, x798, x799, x800, x801, x802, x803, x804, x805,
          x806, x807, x808, x809, x810},
      Val100Array{x811, x812, x813, x814, x815, x816, x817, x818, x819, x820, x821, x822, x823,
                  x824, x825, x826, x827, x828, x829, x830, x831, x832, x833, x834, x835, x836,
                  x837, x838, x839, x840, x841, x842, x843, x844, x845, x846, x847, x848, x849,
                  x850, x851, x852, x853, x854, x855, x856, x857, x858, x859, x860, x861, x862,
                  x863, x864, x865, x866, x867, x868, x869, x870, x871, x872, x873, x874, x875,
                  x876, x877, x878, x879, x880, x881, x882, x883, x884, x885, x886, x887, x888,
                  x889, x890, x891, x892, x893, x894, x895, x896, x897, x898, x899, x900, x901,
                  x902, x903, x904, x905, x906, x907, x908, x909, x910},
      x8,
      LAYOUT_LOOKUP(layout1, _super));
  return x911;
}

} // namespace risc0::impl
