#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

<<<<<<< HEAD
__device__ void step_Top_10(ExecContext& ctx, MutableBuf arg0) {
  // ControlState(zirgen/circuit/keccak2/top.zir:403)
  // ComputeCurrentStep(zirgen/circuit/keccak2/top.zir:460)
  // Top(zirgen/circuit/keccak2/top.zir:483)
  set(ctx, arg0, 12, Val(8));
  // ControlState(zirgen/circuit/keccak2/top.zir:404)
  set(ctx, arg0, 13, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:405)
  set(ctx, arg0, 14, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:406)
  set(ctx, arg0, 15, get(ctx, arg0, 15, 1));
  return;
}
__device__ void step_Top_26(ExecContext& ctx, MutableBuf arg0) {
  // TopState(zirgen/circuit/keccak2/top.zir:43)
  // Top(zirgen/circuit/keccak2/top.zir:493)
  Val x1 = get(ctx, arg0, 916, 1);
  Val x2 = get(ctx, arg0, 917, 1);
  Val x3 = get(ctx, arg0, 918, 1);
  Val x4 = get(ctx, arg0, 919, 1);
  Val x5 = get(ctx, arg0, 920, 1);
  Val x6 = get(ctx, arg0, 921, 1);
  Val x7 = get(ctx, arg0, 922, 1);
  Val x8 = get(ctx, arg0, 923, 1);
  Val x9 = get(ctx, arg0, 924, 1);
  Val x10 = get(ctx, arg0, 925, 1);
  Val x11 = get(ctx, arg0, 926, 1);
  Val x12 = get(ctx, arg0, 927, 1);
  Val x13 = get(ctx, arg0, 928, 1);
  Val x14 = get(ctx, arg0, 929, 1);
  Val x15 = get(ctx, arg0, 930, 1);
  Val x16 = get(ctx, arg0, 931, 1);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  // ReadCycle(zirgen/circuit/keccak2/top.zir:333)
  Val x17 = INVOKE_EXTERN(ctx, getPreimage, Val(0));
  set(ctx, arg0, 816, x17);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x18 = INVOKE_EXTERN(ctx, getPreimage, Val(1));
  set(ctx, arg0, 817, x18);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x19 = INVOKE_EXTERN(ctx, getPreimage, Val(2));
  set(ctx, arg0, 818, x19);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x20 = INVOKE_EXTERN(ctx, getPreimage, Val(3));
  set(ctx, arg0, 819, x20);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x21 = INVOKE_EXTERN(ctx, getPreimage, Val(4));
  set(ctx, arg0, 820, x21);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x22 = INVOKE_EXTERN(ctx, getPreimage, Val(5));
  set(ctx, arg0, 821, x22);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x23 = INVOKE_EXTERN(ctx, getPreimage, Val(6));
  set(ctx, arg0, 822, x23);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x24 = INVOKE_EXTERN(ctx, getPreimage, Val(7));
  set(ctx, arg0, 823, x24);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x25 = INVOKE_EXTERN(ctx, getPreimage, Val(8));
  set(ctx, arg0, 824, x25);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x26 = INVOKE_EXTERN(ctx, getPreimage, Val(9));
  set(ctx, arg0, 825, x26);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x27 = INVOKE_EXTERN(ctx, getPreimage, Val(10));
  set(ctx, arg0, 826, x27);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x28 = INVOKE_EXTERN(ctx, getPreimage, Val(11));
  set(ctx, arg0, 827, x28);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x29 = INVOKE_EXTERN(ctx, getPreimage, Val(12));
  set(ctx, arg0, 828, x29);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x30 = INVOKE_EXTERN(ctx, getPreimage, Val(13));
  set(ctx, arg0, 829, x30);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x31 = INVOKE_EXTERN(ctx, getPreimage, Val(14));
  set(ctx, arg0, 830, x31);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x32 = INVOKE_EXTERN(ctx, getPreimage, Val(15));
  set(ctx, arg0, 831, x32);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x33 = INVOKE_EXTERN(ctx, getPreimage, Val(16));
  set(ctx, arg0, 832, x33);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x34 = INVOKE_EXTERN(ctx, getPreimage, Val(17));
  set(ctx, arg0, 833, x34);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x35 = INVOKE_EXTERN(ctx, getPreimage, Val(18));
  set(ctx, arg0, 834, x35);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x36 = INVOKE_EXTERN(ctx, getPreimage, Val(19));
  set(ctx, arg0, 835, x36);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x37 = INVOKE_EXTERN(ctx, getPreimage, Val(20));
  set(ctx, arg0, 836, x37);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x38 = INVOKE_EXTERN(ctx, getPreimage, Val(21));
  set(ctx, arg0, 837, x38);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x39 = INVOKE_EXTERN(ctx, getPreimage, Val(22));
  set(ctx, arg0, 838, x39);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x40 = INVOKE_EXTERN(ctx, getPreimage, Val(23));
  set(ctx, arg0, 839, x40);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x41 = INVOKE_EXTERN(ctx, getPreimage, Val(24));
  set(ctx, arg0, 840, x41);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x42 = INVOKE_EXTERN(ctx, getPreimage, Val(25));
  set(ctx, arg0, 841, x42);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x43 = INVOKE_EXTERN(ctx, getPreimage, Val(26));
  set(ctx, arg0, 842, x43);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x44 = INVOKE_EXTERN(ctx, getPreimage, Val(27));
  set(ctx, arg0, 843, x44);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x45 = INVOKE_EXTERN(ctx, getPreimage, Val(28));
  set(ctx, arg0, 844, x45);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x46 = INVOKE_EXTERN(ctx, getPreimage, Val(29));
  set(ctx, arg0, 845, x46);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x47 = INVOKE_EXTERN(ctx, getPreimage, Val(30));
  set(ctx, arg0, 846, x47);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x48 = INVOKE_EXTERN(ctx, getPreimage, Val(31));
  set(ctx, arg0, 847, x48);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x49 = INVOKE_EXTERN(ctx, getPreimage, Val(32));
  set(ctx, arg0, 848, x49);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x50 = INVOKE_EXTERN(ctx, getPreimage, Val(33));
  set(ctx, arg0, 849, x50);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x51 = INVOKE_EXTERN(ctx, getPreimage, Val(34));
  set(ctx, arg0, 850, x51);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x52 = INVOKE_EXTERN(ctx, getPreimage, Val(35));
  set(ctx, arg0, 851, x52);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x53 = INVOKE_EXTERN(ctx, getPreimage, Val(36));
  set(ctx, arg0, 852, x53);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x54 = INVOKE_EXTERN(ctx, getPreimage, Val(37));
  set(ctx, arg0, 853, x54);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x55 = INVOKE_EXTERN(ctx, getPreimage, Val(38));
  set(ctx, arg0, 854, x55);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x56 = INVOKE_EXTERN(ctx, getPreimage, Val(39));
  set(ctx, arg0, 855, x56);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x57 = INVOKE_EXTERN(ctx, getPreimage, Val(40));
  set(ctx, arg0, 856, x57);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x58 = INVOKE_EXTERN(ctx, getPreimage, Val(41));
  set(ctx, arg0, 857, x58);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x59 = INVOKE_EXTERN(ctx, getPreimage, Val(42));
  set(ctx, arg0, 858, x59);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x60 = INVOKE_EXTERN(ctx, getPreimage, Val(43));
  set(ctx, arg0, 859, x60);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x61 = INVOKE_EXTERN(ctx, getPreimage, Val(44));
  set(ctx, arg0, 860, x61);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x62 = INVOKE_EXTERN(ctx, getPreimage, Val(45));
  set(ctx, arg0, 861, x62);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x63 = INVOKE_EXTERN(ctx, getPreimage, Val(46));
  set(ctx, arg0, 862, x63);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x64 = INVOKE_EXTERN(ctx, getPreimage, Val(47));
  set(ctx, arg0, 863, x64);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x65 = INVOKE_EXTERN(ctx, getPreimage, Val(48));
  set(ctx, arg0, 864, x65);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x66 = INVOKE_EXTERN(ctx, getPreimage, Val(49));
  set(ctx, arg0, 865, x66);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x67 = INVOKE_EXTERN(ctx, getPreimage, Val(50));
  set(ctx, arg0, 866, x67);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x68 = INVOKE_EXTERN(ctx, getPreimage, Val(51));
  set(ctx, arg0, 867, x68);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x69 = INVOKE_EXTERN(ctx, getPreimage, Val(52));
  set(ctx, arg0, 868, x69);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x70 = INVOKE_EXTERN(ctx, getPreimage, Val(53));
  set(ctx, arg0, 869, x70);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x71 = INVOKE_EXTERN(ctx, getPreimage, Val(54));
  set(ctx, arg0, 870, x71);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x72 = INVOKE_EXTERN(ctx, getPreimage, Val(55));
  set(ctx, arg0, 871, x72);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x73 = INVOKE_EXTERN(ctx, getPreimage, Val(56));
  set(ctx, arg0, 872, x73);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x74 = INVOKE_EXTERN(ctx, getPreimage, Val(57));
  set(ctx, arg0, 873, x74);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x75 = INVOKE_EXTERN(ctx, getPreimage, Val(58));
  set(ctx, arg0, 874, x75);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x76 = INVOKE_EXTERN(ctx, getPreimage, Val(59));
  set(ctx, arg0, 875, x76);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x77 = INVOKE_EXTERN(ctx, getPreimage, Val(60));
  set(ctx, arg0, 876, x77);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x78 = INVOKE_EXTERN(ctx, getPreimage, Val(61));
  set(ctx, arg0, 877, x78);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x79 = INVOKE_EXTERN(ctx, getPreimage, Val(62));
  set(ctx, arg0, 878, x79);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x80 = INVOKE_EXTERN(ctx, getPreimage, Val(63));
  set(ctx, arg0, 879, x80);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x81 = INVOKE_EXTERN(ctx, getPreimage, Val(64));
  set(ctx, arg0, 880, x81);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x82 = INVOKE_EXTERN(ctx, getPreimage, Val(65));
  set(ctx, arg0, 881, x82);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x83 = INVOKE_EXTERN(ctx, getPreimage, Val(66));
  set(ctx, arg0, 882, x83);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x84 = INVOKE_EXTERN(ctx, getPreimage, Val(67));
  set(ctx, arg0, 883, x84);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x85 = INVOKE_EXTERN(ctx, getPreimage, Val(68));
  set(ctx, arg0, 884, x85);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x86 = INVOKE_EXTERN(ctx, getPreimage, Val(69));
  set(ctx, arg0, 885, x86);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x87 = INVOKE_EXTERN(ctx, getPreimage, Val(70));
  set(ctx, arg0, 886, x87);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x88 = INVOKE_EXTERN(ctx, getPreimage, Val(71));
  set(ctx, arg0, 887, x88);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x89 = INVOKE_EXTERN(ctx, getPreimage, Val(72));
  set(ctx, arg0, 888, x89);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x90 = INVOKE_EXTERN(ctx, getPreimage, Val(73));
  set(ctx, arg0, 889, x90);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x91 = INVOKE_EXTERN(ctx, getPreimage, Val(74));
  set(ctx, arg0, 890, x91);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x92 = INVOKE_EXTERN(ctx, getPreimage, Val(75));
  set(ctx, arg0, 891, x92);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x93 = INVOKE_EXTERN(ctx, getPreimage, Val(76));
  set(ctx, arg0, 892, x93);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x94 = INVOKE_EXTERN(ctx, getPreimage, Val(77));
  set(ctx, arg0, 893, x94);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x95 = INVOKE_EXTERN(ctx, getPreimage, Val(78));
  set(ctx, arg0, 894, x95);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x96 = INVOKE_EXTERN(ctx, getPreimage, Val(79));
  set(ctx, arg0, 895, x96);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x97 = INVOKE_EXTERN(ctx, getPreimage, Val(80));
  set(ctx, arg0, 896, x97);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x98 = INVOKE_EXTERN(ctx, getPreimage, Val(81));
  set(ctx, arg0, 897, x98);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x99 = INVOKE_EXTERN(ctx, getPreimage, Val(82));
  set(ctx, arg0, 898, x99);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x100 = INVOKE_EXTERN(ctx, getPreimage, Val(83));
  set(ctx, arg0, 899, x100);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x101 = INVOKE_EXTERN(ctx, getPreimage, Val(84));
  set(ctx, arg0, 900, x101);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x102 = INVOKE_EXTERN(ctx, getPreimage, Val(85));
  set(ctx, arg0, 901, x102);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x103 = INVOKE_EXTERN(ctx, getPreimage, Val(86));
  set(ctx, arg0, 902, x103);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x104 = INVOKE_EXTERN(ctx, getPreimage, Val(87));
  set(ctx, arg0, 903, x104);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x105 = INVOKE_EXTERN(ctx, getPreimage, Val(88));
  set(ctx, arg0, 904, x105);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x106 = INVOKE_EXTERN(ctx, getPreimage, Val(89));
  set(ctx, arg0, 905, x106);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x107 = INVOKE_EXTERN(ctx, getPreimage, Val(90));
  set(ctx, arg0, 906, x107);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x108 = INVOKE_EXTERN(ctx, getPreimage, Val(91));
  set(ctx, arg0, 907, x108);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x109 = INVOKE_EXTERN(ctx, getPreimage, Val(92));
  set(ctx, arg0, 908, x109);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x110 = INVOKE_EXTERN(ctx, getPreimage, Val(93));
  set(ctx, arg0, 909, x110);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x111 = INVOKE_EXTERN(ctx, getPreimage, Val(94));
  set(ctx, arg0, 910, x111);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x112 = INVOKE_EXTERN(ctx, getPreimage, Val(95));
  set(ctx, arg0, 911, x112);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x113 = INVOKE_EXTERN(ctx, getPreimage, Val(96));
  set(ctx, arg0, 912, x113);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x114 = INVOKE_EXTERN(ctx, getPreimage, Val(97));
  set(ctx, arg0, 913, x114);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x115 = INVOKE_EXTERN(ctx, getPreimage, Val(98));
  set(ctx, arg0, 914, x115);
  // GetPreimage(zirgen/circuit/keccak2/top.zir:279)
  Val x116 = INVOKE_EXTERN(ctx, getPreimage, Val(99));
  set(ctx, arg0, 915, x116);
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // SFlatToBits(zirgen/circuit/keccak2/top.zir:329)
  // ReadCycle(zirgen/circuit/keccak2/top.zir:335)
  set(ctx, arg0, 16, Val(0));
  set(ctx, arg0, 17, Val(0));
  set(ctx, arg0, 18, Val(0));
  set(ctx, arg0, 19, Val(0));
  set(ctx, arg0, 20, Val(0));
  set(ctx, arg0, 21, Val(0));
  set(ctx, arg0, 22, Val(0));
  set(ctx, arg0, 23, Val(0));
  set(ctx, arg0, 24, Val(0));
  set(ctx, arg0, 25, Val(0));
  set(ctx, arg0, 26, Val(0));
  set(ctx, arg0, 27, Val(0));
  set(ctx, arg0, 28, Val(0));
  set(ctx, arg0, 29, Val(0));
  set(ctx, arg0, 30, Val(0));
  set(ctx, arg0, 31, Val(0));
  set(ctx, arg0, 32, Val(0));
  set(ctx, arg0, 33, Val(0));
  set(ctx, arg0, 34, Val(0));
  set(ctx, arg0, 35, Val(0));
  set(ctx, arg0, 36, Val(0));
  set(ctx, arg0, 37, Val(0));
  set(ctx, arg0, 38, Val(0));
  set(ctx, arg0, 39, Val(0));
  set(ctx, arg0, 40, Val(0));
  set(ctx, arg0, 41, Val(0));
  set(ctx, arg0, 42, Val(0));
  set(ctx, arg0, 43, Val(0));
  set(ctx, arg0, 44, Val(0));
  set(ctx, arg0, 45, Val(0));
  set(ctx, arg0, 46, Val(0));
  set(ctx, arg0, 47, Val(0));
  set(ctx, arg0, 48, Val(0));
  set(ctx, arg0, 49, Val(0));
  set(ctx, arg0, 50, Val(0));
  set(ctx, arg0, 51, Val(0));
  set(ctx, arg0, 52, Val(0));
  set(ctx, arg0, 53, Val(0));
  set(ctx, arg0, 54, Val(0));
  set(ctx, arg0, 55, Val(0));
  set(ctx, arg0, 56, Val(0));
  set(ctx, arg0, 57, Val(0));
  set(ctx, arg0, 58, Val(0));
  set(ctx, arg0, 59, Val(0));
  set(ctx, arg0, 60, Val(0));
  set(ctx, arg0, 61, Val(0));
  set(ctx, arg0, 62, Val(0));
  set(ctx, arg0, 63, Val(0));
  set(ctx, arg0, 64, Val(0));
  set(ctx, arg0, 65, Val(0));
  set(ctx, arg0, 66, Val(0));
  set(ctx, arg0, 67, Val(0));
  set(ctx, arg0, 68, Val(0));
  set(ctx, arg0, 69, Val(0));
  set(ctx, arg0, 70, Val(0));
  set(ctx, arg0, 71, Val(0));
  set(ctx, arg0, 72, Val(0));
  set(ctx, arg0, 73, Val(0));
  set(ctx, arg0, 74, Val(0));
  set(ctx, arg0, 75, Val(0));
  set(ctx, arg0, 76, Val(0));
  set(ctx, arg0, 77, Val(0));
  set(ctx, arg0, 78, Val(0));
  set(ctx, arg0, 79, Val(0));
  set(ctx, arg0, 80, Val(0));
  set(ctx, arg0, 81, Val(0));
  set(ctx, arg0, 82, Val(0));
  set(ctx, arg0, 83, Val(0));
  set(ctx, arg0, 84, Val(0));
  set(ctx, arg0, 85, Val(0));
  set(ctx, arg0, 86, Val(0));
  set(ctx, arg0, 87, Val(0));
  set(ctx, arg0, 88, Val(0));
  set(ctx, arg0, 89, Val(0));
  set(ctx, arg0, 90, Val(0));
  set(ctx, arg0, 91, Val(0));
  set(ctx, arg0, 92, Val(0));
  set(ctx, arg0, 93, Val(0));
  set(ctx, arg0, 94, Val(0));
  set(ctx, arg0, 95, Val(0));
  set(ctx, arg0, 96, Val(0));
  set(ctx, arg0, 97, Val(0));
  set(ctx, arg0, 98, Val(0));
  set(ctx, arg0, 99, Val(0));
  set(ctx, arg0, 100, Val(0));
  set(ctx, arg0, 101, Val(0));
  set(ctx, arg0, 102, Val(0));
  set(ctx, arg0, 103, Val(0));
  set(ctx, arg0, 104, Val(0));
  set(ctx, arg0, 105, Val(0));
  set(ctx, arg0, 106, Val(0));
  set(ctx, arg0, 107, Val(0));
  set(ctx, arg0, 108, Val(0));
  set(ctx, arg0, 109, Val(0));
  set(ctx, arg0, 110, Val(0));
  set(ctx, arg0, 111, Val(0));
  set(ctx, arg0, 112, Val(0));
  set(ctx, arg0, 113, Val(0));
  set(ctx, arg0, 114, Val(0));
  set(ctx, arg0, 115, Val(0));
  set(ctx, arg0, 116, Val(0));
  set(ctx, arg0, 117, Val(0));
  set(ctx, arg0, 118, Val(0));
  set(ctx, arg0, 119, Val(0));
  set(ctx, arg0, 120, Val(0));
  set(ctx, arg0, 121, Val(0));
  set(ctx, arg0, 122, Val(0));
  set(ctx, arg0, 123, Val(0));
  set(ctx, arg0, 124, Val(0));
  set(ctx, arg0, 125, Val(0));
  set(ctx, arg0, 126, Val(0));
  set(ctx, arg0, 127, Val(0));
  set(ctx, arg0, 128, Val(0));
  set(ctx, arg0, 129, Val(0));
  set(ctx, arg0, 130, Val(0));
  set(ctx, arg0, 131, Val(0));
  set(ctx, arg0, 132, Val(0));
  set(ctx, arg0, 133, Val(0));
  set(ctx, arg0, 134, Val(0));
  set(ctx, arg0, 135, Val(0));
  set(ctx, arg0, 136, Val(0));
  set(ctx, arg0, 137, Val(0));
  set(ctx, arg0, 138, Val(0));
  set(ctx, arg0, 139, Val(0));
  set(ctx, arg0, 140, Val(0));
  set(ctx, arg0, 141, Val(0));
  set(ctx, arg0, 142, Val(0));
  set(ctx, arg0, 143, Val(0));
  set(ctx, arg0, 144, bitAnd(x7, Val(1)));
  set(ctx, arg0, 145, (bitAnd(x7, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 146, (bitAnd(x7, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 147, (bitAnd(x7, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 148, (bitAnd(x7, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 149, (bitAnd(x7, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 150, (bitAnd(x7, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 151, (bitAnd(x7, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 152, (bitAnd(x7, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 153, (bitAnd(x7, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 154, (bitAnd(x7, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 155, (bitAnd(x7, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 156, (bitAnd(x7, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 157, (bitAnd(x7, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 158, (bitAnd(x7, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 159, (bitAnd(x7, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 160, bitAnd(x8, Val(1)));
  set(ctx, arg0, 161, (bitAnd(x8, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 162, (bitAnd(x8, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 163, (bitAnd(x8, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 164, (bitAnd(x8, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 165, (bitAnd(x8, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 166, (bitAnd(x8, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 167, (bitAnd(x8, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 168, (bitAnd(x8, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 169, (bitAnd(x8, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 170, (bitAnd(x8, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 171, (bitAnd(x8, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 172, (bitAnd(x8, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 173, (bitAnd(x8, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 174, (bitAnd(x8, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 175, (bitAnd(x8, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 176, bitAnd(x5, Val(1)));
  set(ctx, arg0, 177, (bitAnd(x5, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 178, (bitAnd(x5, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 179, (bitAnd(x5, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 180, (bitAnd(x5, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 181, (bitAnd(x5, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 182, (bitAnd(x5, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 183, (bitAnd(x5, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 184, (bitAnd(x5, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 185, (bitAnd(x5, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 186, (bitAnd(x5, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 187, (bitAnd(x5, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 188, (bitAnd(x5, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 189, (bitAnd(x5, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 190, (bitAnd(x5, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 191, (bitAnd(x5, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 192, bitAnd(x6, Val(1)));
  set(ctx, arg0, 193, (bitAnd(x6, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 194, (bitAnd(x6, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 195, (bitAnd(x6, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 196, (bitAnd(x6, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 197, (bitAnd(x6, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 198, (bitAnd(x6, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 199, (bitAnd(x6, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 200, (bitAnd(x6, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 201, (bitAnd(x6, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 202, (bitAnd(x6, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 203, (bitAnd(x6, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 204, (bitAnd(x6, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 205, (bitAnd(x6, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 206, (bitAnd(x6, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 207, (bitAnd(x6, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 208, bitAnd(x3, Val(1)));
  set(ctx, arg0, 209, (bitAnd(x3, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 210, (bitAnd(x3, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 211, (bitAnd(x3, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 212, (bitAnd(x3, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 213, (bitAnd(x3, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 214, (bitAnd(x3, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 215, (bitAnd(x3, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 216, (bitAnd(x3, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 217, (bitAnd(x3, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 218, (bitAnd(x3, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 219, (bitAnd(x3, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 220, (bitAnd(x3, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 221, (bitAnd(x3, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 222, (bitAnd(x3, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 223, (bitAnd(x3, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 224, bitAnd(x4, Val(1)));
  set(ctx, arg0, 225, (bitAnd(x4, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 226, (bitAnd(x4, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 227, (bitAnd(x4, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 228, (bitAnd(x4, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 229, (bitAnd(x4, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 230, (bitAnd(x4, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 231, (bitAnd(x4, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 232, (bitAnd(x4, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 233, (bitAnd(x4, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 234, (bitAnd(x4, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 235, (bitAnd(x4, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 236, (bitAnd(x4, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 237, (bitAnd(x4, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 238, (bitAnd(x4, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 239, (bitAnd(x4, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 240, bitAnd(x1, Val(1)));
  set(ctx, arg0, 241, (bitAnd(x1, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 242, (bitAnd(x1, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 243, (bitAnd(x1, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 244, (bitAnd(x1, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 245, (bitAnd(x1, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 246, (bitAnd(x1, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 247, (bitAnd(x1, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 248, (bitAnd(x1, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 249, (bitAnd(x1, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 250, (bitAnd(x1, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 251, (bitAnd(x1, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 252, (bitAnd(x1, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 253, (bitAnd(x1, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 254, (bitAnd(x1, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 255, (bitAnd(x1, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 256, bitAnd(x2, Val(1)));
  set(ctx, arg0, 257, (bitAnd(x2, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 258, (bitAnd(x2, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 259, (bitAnd(x2, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 260, (bitAnd(x2, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 261, (bitAnd(x2, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 262, (bitAnd(x2, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 263, (bitAnd(x2, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 264, (bitAnd(x2, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 265, (bitAnd(x2, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 266, (bitAnd(x2, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 267, (bitAnd(x2, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 268, (bitAnd(x2, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 269, (bitAnd(x2, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 270, (bitAnd(x2, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 271, (bitAnd(x2, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 272, Val(0));
  set(ctx, arg0, 273, Val(0));
  set(ctx, arg0, 274, Val(0));
  set(ctx, arg0, 275, Val(0));
  set(ctx, arg0, 276, Val(0));
  set(ctx, arg0, 277, Val(0));
  set(ctx, arg0, 278, Val(0));
  set(ctx, arg0, 279, Val(0));
  set(ctx, arg0, 280, Val(0));
  set(ctx, arg0, 281, Val(0));
  set(ctx, arg0, 282, Val(0));
  set(ctx, arg0, 283, Val(0));
  set(ctx, arg0, 284, Val(0));
  set(ctx, arg0, 285, Val(0));
  set(ctx, arg0, 286, Val(0));
  set(ctx, arg0, 287, Val(0));
  set(ctx, arg0, 288, Val(0));
  set(ctx, arg0, 289, Val(0));
  set(ctx, arg0, 290, Val(0));
  set(ctx, arg0, 291, Val(0));
  set(ctx, arg0, 292, Val(0));
  set(ctx, arg0, 293, Val(0));
  set(ctx, arg0, 294, Val(0));
  set(ctx, arg0, 295, Val(0));
  set(ctx, arg0, 296, Val(0));
  set(ctx, arg0, 297, Val(0));
  set(ctx, arg0, 298, Val(0));
  set(ctx, arg0, 299, Val(0));
  set(ctx, arg0, 300, Val(0));
  set(ctx, arg0, 301, Val(0));
  set(ctx, arg0, 302, Val(0));
  set(ctx, arg0, 303, Val(0));
  set(ctx, arg0, 304, Val(0));
  set(ctx, arg0, 305, Val(0));
  set(ctx, arg0, 306, Val(0));
  set(ctx, arg0, 307, Val(0));
  set(ctx, arg0, 308, Val(0));
  set(ctx, arg0, 309, Val(0));
  set(ctx, arg0, 310, Val(0));
  set(ctx, arg0, 311, Val(0));
  set(ctx, arg0, 312, Val(0));
  set(ctx, arg0, 313, Val(0));
  set(ctx, arg0, 314, Val(0));
  set(ctx, arg0, 315, Val(0));
  set(ctx, arg0, 316, Val(0));
  set(ctx, arg0, 317, Val(0));
  set(ctx, arg0, 318, Val(0));
  set(ctx, arg0, 319, Val(0));
  set(ctx, arg0, 320, Val(0));
  set(ctx, arg0, 321, Val(0));
  set(ctx, arg0, 322, Val(0));
  set(ctx, arg0, 323, Val(0));
  set(ctx, arg0, 324, Val(0));
  set(ctx, arg0, 325, Val(0));
  set(ctx, arg0, 326, Val(0));
  set(ctx, arg0, 327, Val(0));
  set(ctx, arg0, 328, Val(0));
  set(ctx, arg0, 329, Val(0));
  set(ctx, arg0, 330, Val(0));
  set(ctx, arg0, 331, Val(0));
  set(ctx, arg0, 332, Val(0));
  set(ctx, arg0, 333, Val(0));
  set(ctx, arg0, 334, Val(0));
  set(ctx, arg0, 335, Val(0));
  set(ctx, arg0, 336, Val(0));
  set(ctx, arg0, 337, Val(0));
  set(ctx, arg0, 338, Val(0));
  set(ctx, arg0, 339, Val(0));
  set(ctx, arg0, 340, Val(0));
  set(ctx, arg0, 341, Val(0));
  set(ctx, arg0, 342, Val(0));
  set(ctx, arg0, 343, Val(0));
  set(ctx, arg0, 344, Val(0));
  set(ctx, arg0, 345, Val(0));
  set(ctx, arg0, 346, Val(0));
  set(ctx, arg0, 347, Val(0));
  set(ctx, arg0, 348, Val(0));
  set(ctx, arg0, 349, Val(0));
  set(ctx, arg0, 350, Val(0));
  set(ctx, arg0, 351, Val(0));
  set(ctx, arg0, 352, Val(0));
  set(ctx, arg0, 353, Val(0));
  set(ctx, arg0, 354, Val(0));
  set(ctx, arg0, 355, Val(0));
  set(ctx, arg0, 356, Val(0));
  set(ctx, arg0, 357, Val(0));
  set(ctx, arg0, 358, Val(0));
  set(ctx, arg0, 359, Val(0));
  set(ctx, arg0, 360, Val(0));
  set(ctx, arg0, 361, Val(0));
  set(ctx, arg0, 362, Val(0));
  set(ctx, arg0, 363, Val(0));
  set(ctx, arg0, 364, Val(0));
  set(ctx, arg0, 365, Val(0));
  set(ctx, arg0, 366, Val(0));
  set(ctx, arg0, 367, Val(0));
  set(ctx, arg0, 368, Val(0));
  set(ctx, arg0, 369, Val(0));
  set(ctx, arg0, 370, Val(0));
  set(ctx, arg0, 371, Val(0));
  set(ctx, arg0, 372, Val(0));
  set(ctx, arg0, 373, Val(0));
  set(ctx, arg0, 374, Val(0));
  set(ctx, arg0, 375, Val(0));
  set(ctx, arg0, 376, Val(0));
  set(ctx, arg0, 377, Val(0));
  set(ctx, arg0, 378, Val(0));
  set(ctx, arg0, 379, Val(0));
  set(ctx, arg0, 380, Val(0));
  set(ctx, arg0, 381, Val(0));
  set(ctx, arg0, 382, Val(0));
  set(ctx, arg0, 383, Val(0));
  set(ctx, arg0, 384, Val(0));
  set(ctx, arg0, 385, Val(0));
  set(ctx, arg0, 386, Val(0));
  set(ctx, arg0, 387, Val(0));
  set(ctx, arg0, 388, Val(0));
  set(ctx, arg0, 389, Val(0));
  set(ctx, arg0, 390, Val(0));
  set(ctx, arg0, 391, Val(0));
  set(ctx, arg0, 392, Val(0));
  set(ctx, arg0, 393, Val(0));
  set(ctx, arg0, 394, Val(0));
  set(ctx, arg0, 395, Val(0));
  set(ctx, arg0, 396, Val(0));
  set(ctx, arg0, 397, Val(0));
  set(ctx, arg0, 398, Val(0));
  set(ctx, arg0, 399, Val(0));
  set(ctx, arg0, 400, bitAnd(x15, Val(1)));
  set(ctx, arg0, 401, (bitAnd(x15, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 402, (bitAnd(x15, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 403, (bitAnd(x15, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 404, (bitAnd(x15, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 405, (bitAnd(x15, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 406, (bitAnd(x15, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 407, (bitAnd(x15, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 408, (bitAnd(x15, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 409, (bitAnd(x15, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 410, (bitAnd(x15, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 411, (bitAnd(x15, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 412, (bitAnd(x15, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 413, (bitAnd(x15, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 414, (bitAnd(x15, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 415, (bitAnd(x15, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 416, bitAnd(x16, Val(1)));
  set(ctx, arg0, 417, (bitAnd(x16, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 418, (bitAnd(x16, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 419, (bitAnd(x16, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 420, (bitAnd(x16, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 421, (bitAnd(x16, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 422, (bitAnd(x16, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 423, (bitAnd(x16, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 424, (bitAnd(x16, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 425, (bitAnd(x16, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 426, (bitAnd(x16, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 427, (bitAnd(x16, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 428, (bitAnd(x16, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 429, (bitAnd(x16, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 430, (bitAnd(x16, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 431, (bitAnd(x16, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 432, bitAnd(x13, Val(1)));
  set(ctx, arg0, 433, (bitAnd(x13, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 434, (bitAnd(x13, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 435, (bitAnd(x13, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 436, (bitAnd(x13, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 437, (bitAnd(x13, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 438, (bitAnd(x13, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 439, (bitAnd(x13, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 440, (bitAnd(x13, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 441, (bitAnd(x13, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 442, (bitAnd(x13, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 443, (bitAnd(x13, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 444, (bitAnd(x13, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 445, (bitAnd(x13, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 446, (bitAnd(x13, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 447, (bitAnd(x13, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 448, bitAnd(x14, Val(1)));
  set(ctx, arg0, 449, (bitAnd(x14, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 450, (bitAnd(x14, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 451, (bitAnd(x14, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 452, (bitAnd(x14, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 453, (bitAnd(x14, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 454, (bitAnd(x14, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 455, (bitAnd(x14, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 456, (bitAnd(x14, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 457, (bitAnd(x14, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 458, (bitAnd(x14, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 459, (bitAnd(x14, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 460, (bitAnd(x14, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 461, (bitAnd(x14, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 462, (bitAnd(x14, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 463, (bitAnd(x14, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 464, bitAnd(x11, Val(1)));
  set(ctx, arg0, 465, (bitAnd(x11, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 466, (bitAnd(x11, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 467, (bitAnd(x11, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 468, (bitAnd(x11, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 469, (bitAnd(x11, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 470, (bitAnd(x11, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 471, (bitAnd(x11, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 472, (bitAnd(x11, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 473, (bitAnd(x11, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 474, (bitAnd(x11, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 475, (bitAnd(x11, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 476, (bitAnd(x11, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 477, (bitAnd(x11, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 478, (bitAnd(x11, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 479, (bitAnd(x11, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 480, bitAnd(x12, Val(1)));
  set(ctx, arg0, 481, (bitAnd(x12, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 482, (bitAnd(x12, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 483, (bitAnd(x12, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 484, (bitAnd(x12, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 485, (bitAnd(x12, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 486, (bitAnd(x12, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 487, (bitAnd(x12, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 488, (bitAnd(x12, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 489, (bitAnd(x12, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 490, (bitAnd(x12, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 491, (bitAnd(x12, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 492, (bitAnd(x12, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 493, (bitAnd(x12, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 494, (bitAnd(x12, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 495, (bitAnd(x12, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 496, bitAnd(x9, Val(1)));
  set(ctx, arg0, 497, (bitAnd(x9, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 498, (bitAnd(x9, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 499, (bitAnd(x9, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 500, (bitAnd(x9, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 501, (bitAnd(x9, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 502, (bitAnd(x9, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 503, (bitAnd(x9, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 504, (bitAnd(x9, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 505, (bitAnd(x9, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 506, (bitAnd(x9, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 507, (bitAnd(x9, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 508, (bitAnd(x9, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 509, (bitAnd(x9, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 510, (bitAnd(x9, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 511, (bitAnd(x9, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 512, bitAnd(x10, Val(1)));
  set(ctx, arg0, 513, (bitAnd(x10, Val(2)) * Val(1006632961)));
  set(ctx, arg0, 514, (bitAnd(x10, Val(4)) * Val(1509949441)));
  set(ctx, arg0, 515, (bitAnd(x10, Val(8)) * Val(1761607681)));
  set(ctx, arg0, 516, (bitAnd(x10, Val(16)) * Val(1887436801)));
  set(ctx, arg0, 517, (bitAnd(x10, Val(32)) * Val(1950351361)));
  set(ctx, arg0, 518, (bitAnd(x10, Val(64)) * Val(1981808641)));
  set(ctx, arg0, 519, (bitAnd(x10, Val(128)) * Val(1997537281)));
  set(ctx, arg0, 520, (bitAnd(x10, Val(256)) * Val(2005401601)));
  set(ctx, arg0, 521, (bitAnd(x10, Val(512)) * Val(2009333761)));
  set(ctx, arg0, 522, (bitAnd(x10, Val(1024)) * Val(2011299841)));
  set(ctx, arg0, 523, (bitAnd(x10, Val(2048)) * Val(2012282881)));
  set(ctx, arg0, 524, (bitAnd(x10, Val(4096)) * Val(2012774401)));
  set(ctx, arg0, 525, (bitAnd(x10, Val(8192)) * Val(2013020161)));
  set(ctx, arg0, 526, (bitAnd(x10, Val(16384)) * Val(2013143041)));
  set(ctx, arg0, 527, (bitAnd(x10, Val(32768)) * Val(2013204481)));
  set(ctx, arg0, 528, Val(0));
  set(ctx, arg0, 529, Val(0));
  set(ctx, arg0, 530, Val(0));
  set(ctx, arg0, 531, Val(0));
  set(ctx, arg0, 532, Val(0));
  set(ctx, arg0, 533, Val(0));
  set(ctx, arg0, 534, Val(0));
  set(ctx, arg0, 535, Val(0));
  set(ctx, arg0, 536, Val(0));
  set(ctx, arg0, 537, Val(0));
  set(ctx, arg0, 538, Val(0));
  set(ctx, arg0, 539, Val(0));
  set(ctx, arg0, 540, Val(0));
  set(ctx, arg0, 541, Val(0));
  set(ctx, arg0, 542, Val(0));
  set(ctx, arg0, 543, Val(0));
  set(ctx, arg0, 544, Val(0));
  set(ctx, arg0, 545, Val(0));
  set(ctx, arg0, 546, Val(0));
  set(ctx, arg0, 547, Val(0));
  set(ctx, arg0, 548, Val(0));
  set(ctx, arg0, 549, Val(0));
  set(ctx, arg0, 550, Val(0));
  set(ctx, arg0, 551, Val(0));
  set(ctx, arg0, 552, Val(0));
  set(ctx, arg0, 553, Val(0));
  set(ctx, arg0, 554, Val(0));
  set(ctx, arg0, 555, Val(0));
  set(ctx, arg0, 556, Val(0));
  set(ctx, arg0, 557, Val(0));
  set(ctx, arg0, 558, Val(0));
  set(ctx, arg0, 559, Val(0));
  set(ctx, arg0, 560, Val(0));
  set(ctx, arg0, 561, Val(0));
  set(ctx, arg0, 562, Val(0));
  set(ctx, arg0, 563, Val(0));
  set(ctx, arg0, 564, Val(0));
  set(ctx, arg0, 565, Val(0));
  set(ctx, arg0, 566, Val(0));
  set(ctx, arg0, 567, Val(0));
  set(ctx, arg0, 568, Val(0));
  set(ctx, arg0, 569, Val(0));
  set(ctx, arg0, 570, Val(0));
  set(ctx, arg0, 571, Val(0));
  set(ctx, arg0, 572, Val(0));
  set(ctx, arg0, 573, Val(0));
  set(ctx, arg0, 574, Val(0));
  set(ctx, arg0, 575, Val(0));
  set(ctx, arg0, 576, Val(0));
  set(ctx, arg0, 577, Val(0));
  set(ctx, arg0, 578, Val(0));
  set(ctx, arg0, 579, Val(0));
  set(ctx, arg0, 580, Val(0));
  set(ctx, arg0, 581, Val(0));
  set(ctx, arg0, 582, Val(0));
  set(ctx, arg0, 583, Val(0));
  set(ctx, arg0, 584, Val(0));
  set(ctx, arg0, 585, Val(0));
  set(ctx, arg0, 586, Val(0));
  set(ctx, arg0, 587, Val(0));
  set(ctx, arg0, 588, Val(0));
  set(ctx, arg0, 589, Val(0));
  set(ctx, arg0, 590, Val(0));
  set(ctx, arg0, 591, Val(0));
  set(ctx, arg0, 592, Val(0));
  set(ctx, arg0, 593, Val(0));
  set(ctx, arg0, 594, Val(0));
  set(ctx, arg0, 595, Val(0));
  set(ctx, arg0, 596, Val(0));
  set(ctx, arg0, 597, Val(0));
  set(ctx, arg0, 598, Val(0));
  set(ctx, arg0, 599, Val(0));
  set(ctx, arg0, 600, Val(0));
  set(ctx, arg0, 601, Val(0));
  set(ctx, arg0, 602, Val(0));
  set(ctx, arg0, 603, Val(0));
  set(ctx, arg0, 604, Val(0));
  set(ctx, arg0, 605, Val(0));
  set(ctx, arg0, 606, Val(0));
  set(ctx, arg0, 607, Val(0));
  set(ctx, arg0, 608, Val(0));
  set(ctx, arg0, 609, Val(0));
  set(ctx, arg0, 610, Val(0));
  set(ctx, arg0, 611, Val(0));
  set(ctx, arg0, 612, Val(0));
  set(ctx, arg0, 613, Val(0));
  set(ctx, arg0, 614, Val(0));
  set(ctx, arg0, 615, Val(0));
  set(ctx, arg0, 616, Val(0));
  set(ctx, arg0, 617, Val(0));
  set(ctx, arg0, 618, Val(0));
  set(ctx, arg0, 619, Val(0));
  set(ctx, arg0, 620, Val(0));
  set(ctx, arg0, 621, Val(0));
  set(ctx, arg0, 622, Val(0));
  set(ctx, arg0, 623, Val(0));
  set(ctx, arg0, 624, Val(0));
  set(ctx, arg0, 625, Val(0));
  set(ctx, arg0, 626, Val(0));
  set(ctx, arg0, 627, Val(0));
  set(ctx, arg0, 628, Val(0));
  set(ctx, arg0, 629, Val(0));
  set(ctx, arg0, 630, Val(0));
  set(ctx, arg0, 631, Val(0));
  set(ctx, arg0, 632, Val(0));
  set(ctx, arg0, 633, Val(0));
  set(ctx, arg0, 634, Val(0));
  set(ctx, arg0, 635, Val(0));
  set(ctx, arg0, 636, Val(0));
  set(ctx, arg0, 637, Val(0));
  set(ctx, arg0, 638, Val(0));
  set(ctx, arg0, 639, Val(0));
  set(ctx, arg0, 640, Val(0));
  set(ctx, arg0, 641, Val(0));
  set(ctx, arg0, 642, Val(0));
  set(ctx, arg0, 643, Val(0));
  set(ctx, arg0, 644, Val(0));
  set(ctx, arg0, 645, Val(0));
  set(ctx, arg0, 646, Val(0));
  set(ctx, arg0, 647, Val(0));
  set(ctx, arg0, 648, Val(0));
  set(ctx, arg0, 649, Val(0));
  set(ctx, arg0, 650, Val(0));
  set(ctx, arg0, 651, Val(0));
  set(ctx, arg0, 652, Val(0));
  set(ctx, arg0, 653, Val(0));
  set(ctx, arg0, 654, Val(0));
  set(ctx, arg0, 655, Val(0));
  set(ctx, arg0, 656, Val(0));
  set(ctx, arg0, 657, Val(0));
  set(ctx, arg0, 658, Val(0));
  set(ctx, arg0, 659, Val(0));
  set(ctx, arg0, 660, Val(0));
  set(ctx, arg0, 661, Val(0));
  set(ctx, arg0, 662, Val(0));
  set(ctx, arg0, 663, Val(0));
  set(ctx, arg0, 664, Val(0));
  set(ctx, arg0, 665, Val(0));
  set(ctx, arg0, 666, Val(0));
  set(ctx, arg0, 667, Val(0));
  set(ctx, arg0, 668, Val(0));
  set(ctx, arg0, 669, Val(0));
  set(ctx, arg0, 670, Val(0));
  set(ctx, arg0, 671, Val(0));
  set(ctx, arg0, 672, Val(0));
  set(ctx, arg0, 673, Val(0));
  set(ctx, arg0, 674, Val(0));
  set(ctx, arg0, 675, Val(0));
  set(ctx, arg0, 676, Val(0));
  set(ctx, arg0, 677, Val(0));
  set(ctx, arg0, 678, Val(0));
  set(ctx, arg0, 679, Val(0));
  set(ctx, arg0, 680, Val(0));
  set(ctx, arg0, 681, Val(0));
  set(ctx, arg0, 682, Val(0));
  set(ctx, arg0, 683, Val(0));
  set(ctx, arg0, 684, Val(0));
  set(ctx, arg0, 685, Val(0));
  set(ctx, arg0, 686, Val(0));
  set(ctx, arg0, 687, Val(0));
  set(ctx, arg0, 688, Val(0));
  set(ctx, arg0, 689, Val(0));
  set(ctx, arg0, 690, Val(0));
  set(ctx, arg0, 691, Val(0));
  set(ctx, arg0, 692, Val(0));
  set(ctx, arg0, 693, Val(0));
  set(ctx, arg0, 694, Val(0));
  set(ctx, arg0, 695, Val(0));
  set(ctx, arg0, 696, Val(0));
  set(ctx, arg0, 697, Val(0));
  set(ctx, arg0, 698, Val(0));
  set(ctx, arg0, 699, Val(0));
  set(ctx, arg0, 700, Val(0));
  set(ctx, arg0, 701, Val(0));
  set(ctx, arg0, 702, Val(0));
  set(ctx, arg0, 703, Val(0));
  set(ctx, arg0, 704, Val(0));
  set(ctx, arg0, 705, Val(0));
  set(ctx, arg0, 706, Val(0));
  set(ctx, arg0, 707, Val(0));
  set(ctx, arg0, 708, Val(0));
  set(ctx, arg0, 709, Val(0));
  set(ctx, arg0, 710, Val(0));
  set(ctx, arg0, 711, Val(0));
  set(ctx, arg0, 712, Val(0));
  set(ctx, arg0, 713, Val(0));
  set(ctx, arg0, 714, Val(0));
  set(ctx, arg0, 715, Val(0));
  set(ctx, arg0, 716, Val(0));
  set(ctx, arg0, 717, Val(0));
  set(ctx, arg0, 718, Val(0));
  set(ctx, arg0, 719, Val(0));
  set(ctx, arg0, 720, Val(0));
  set(ctx, arg0, 721, Val(0));
  set(ctx, arg0, 722, Val(0));
  set(ctx, arg0, 723, Val(0));
  set(ctx, arg0, 724, Val(0));
  set(ctx, arg0, 725, Val(0));
  set(ctx, arg0, 726, Val(0));
  set(ctx, arg0, 727, Val(0));
  set(ctx, arg0, 728, Val(0));
  set(ctx, arg0, 729, Val(0));
  set(ctx, arg0, 730, Val(0));
  set(ctx, arg0, 731, Val(0));
  set(ctx, arg0, 732, Val(0));
  set(ctx, arg0, 733, Val(0));
  set(ctx, arg0, 734, Val(0));
  set(ctx, arg0, 735, Val(0));
  set(ctx, arg0, 736, Val(0));
  set(ctx, arg0, 737, Val(0));
  set(ctx, arg0, 738, Val(0));
  set(ctx, arg0, 739, Val(0));
  set(ctx, arg0, 740, Val(0));
  set(ctx, arg0, 741, Val(0));
  set(ctx, arg0, 742, Val(0));
  set(ctx, arg0, 743, Val(0));
  set(ctx, arg0, 744, Val(0));
  set(ctx, arg0, 745, Val(0));
  set(ctx, arg0, 746, Val(0));
  set(ctx, arg0, 747, Val(0));
  set(ctx, arg0, 748, Val(0));
  set(ctx, arg0, 749, Val(0));
  set(ctx, arg0, 750, Val(0));
  set(ctx, arg0, 751, Val(0));
  set(ctx, arg0, 752, Val(0));
  set(ctx, arg0, 753, Val(0));
  set(ctx, arg0, 754, Val(0));
  set(ctx, arg0, 755, Val(0));
  set(ctx, arg0, 756, Val(0));
  set(ctx, arg0, 757, Val(0));
  set(ctx, arg0, 758, Val(0));
  set(ctx, arg0, 759, Val(0));
  set(ctx, arg0, 760, Val(0));
  set(ctx, arg0, 761, Val(0));
  set(ctx, arg0, 762, Val(0));
  set(ctx, arg0, 763, Val(0));
  set(ctx, arg0, 764, Val(0));
  set(ctx, arg0, 765, Val(0));
  set(ctx, arg0, 766, Val(0));
  set(ctx, arg0, 767, Val(0));
  set(ctx, arg0, 768, Val(0));
  set(ctx, arg0, 769, Val(0));
  set(ctx, arg0, 770, Val(0));
  set(ctx, arg0, 771, Val(0));
  set(ctx, arg0, 772, Val(0));
  set(ctx, arg0, 773, Val(0));
  set(ctx, arg0, 774, Val(0));
  set(ctx, arg0, 775, Val(0));
  set(ctx, arg0, 776, Val(0));
  set(ctx, arg0, 777, Val(0));
  set(ctx, arg0, 778, Val(0));
  set(ctx, arg0, 779, Val(0));
  set(ctx, arg0, 780, Val(0));
  set(ctx, arg0, 781, Val(0));
  set(ctx, arg0, 782, Val(0));
  set(ctx, arg0, 783, Val(0));
  set(ctx, arg0, 784, Val(0));
  set(ctx, arg0, 785, Val(0));
  set(ctx, arg0, 786, Val(0));
  set(ctx, arg0, 787, Val(0));
  set(ctx, arg0, 788, Val(0));
  set(ctx, arg0, 789, Val(0));
  set(ctx, arg0, 790, Val(0));
  set(ctx, arg0, 791, Val(0));
  set(ctx, arg0, 792, Val(0));
  set(ctx, arg0, 793, Val(0));
  set(ctx, arg0, 794, Val(0));
  set(ctx, arg0, 795, Val(0));
  set(ctx, arg0, 796, Val(0));
  set(ctx, arg0, 797, Val(0));
  set(ctx, arg0, 798, Val(0));
  set(ctx, arg0, 799, Val(0));
  set(ctx, arg0, 800, Val(0));
  set(ctx, arg0, 801, Val(0));
  set(ctx, arg0, 802, Val(0));
  set(ctx, arg0, 803, Val(0));
  set(ctx, arg0, 804, Val(0));
  set(ctx, arg0, 805, Val(0));
  set(ctx, arg0, 806, Val(0));
  set(ctx, arg0, 807, Val(0));
  set(ctx, arg0, 808, Val(0));
  set(ctx, arg0, 809, Val(0));
  set(ctx, arg0, 810, Val(0));
  set(ctx, arg0, 811, Val(0));
  set(ctx, arg0, 812, Val(0));
  set(ctx, arg0, 813, Val(0));
  set(ctx, arg0, 814, Val(0));
  set(ctx, arg0, 815, Val(0));
  // Reg(<preamble>:4)
  // TopState(zirgen/circuit/keccak2/top.zir:36)
  // ReadCycle(zirgen/circuit/keccak2/top.zir:336)
  set(ctx, arg0, 16, get(ctx, arg0, 16, 0));
  set(ctx, arg0, 17, get(ctx, arg0, 17, 0));
  set(ctx, arg0, 18, get(ctx, arg0, 18, 0));
  set(ctx, arg0, 19, get(ctx, arg0, 19, 0));
  set(ctx, arg0, 20, get(ctx, arg0, 20, 0));
  set(ctx, arg0, 21, get(ctx, arg0, 21, 0));
  set(ctx, arg0, 22, get(ctx, arg0, 22, 0));
  set(ctx, arg0, 23, get(ctx, arg0, 23, 0));
  set(ctx, arg0, 24, get(ctx, arg0, 24, 0));
  set(ctx, arg0, 25, get(ctx, arg0, 25, 0));
  set(ctx, arg0, 26, get(ctx, arg0, 26, 0));
  set(ctx, arg0, 27, get(ctx, arg0, 27, 0));
  set(ctx, arg0, 28, get(ctx, arg0, 28, 0));
  set(ctx, arg0, 29, get(ctx, arg0, 29, 0));
  set(ctx, arg0, 30, get(ctx, arg0, 30, 0));
  set(ctx, arg0, 31, get(ctx, arg0, 31, 0));
  set(ctx, arg0, 32, get(ctx, arg0, 32, 0));
  set(ctx, arg0, 33, get(ctx, arg0, 33, 0));
  set(ctx, arg0, 34, get(ctx, arg0, 34, 0));
  set(ctx, arg0, 35, get(ctx, arg0, 35, 0));
  set(ctx, arg0, 36, get(ctx, arg0, 36, 0));
  set(ctx, arg0, 37, get(ctx, arg0, 37, 0));
  set(ctx, arg0, 38, get(ctx, arg0, 38, 0));
  set(ctx, arg0, 39, get(ctx, arg0, 39, 0));
  set(ctx, arg0, 40, get(ctx, arg0, 40, 0));
  set(ctx, arg0, 41, get(ctx, arg0, 41, 0));
  set(ctx, arg0, 42, get(ctx, arg0, 42, 0));
  set(ctx, arg0, 43, get(ctx, arg0, 43, 0));
  set(ctx, arg0, 44, get(ctx, arg0, 44, 0));
  set(ctx, arg0, 45, get(ctx, arg0, 45, 0));
  set(ctx, arg0, 46, get(ctx, arg0, 46, 0));
  set(ctx, arg0, 47, get(ctx, arg0, 47, 0));
  set(ctx, arg0, 48, get(ctx, arg0, 48, 0));
  set(ctx, arg0, 49, get(ctx, arg0, 49, 0));
  set(ctx, arg0, 50, get(ctx, arg0, 50, 0));
  set(ctx, arg0, 51, get(ctx, arg0, 51, 0));
  set(ctx, arg0, 52, get(ctx, arg0, 52, 0));
  set(ctx, arg0, 53, get(ctx, arg0, 53, 0));
  set(ctx, arg0, 54, get(ctx, arg0, 54, 0));
  set(ctx, arg0, 55, get(ctx, arg0, 55, 0));
  set(ctx, arg0, 56, get(ctx, arg0, 56, 0));
  set(ctx, arg0, 57, get(ctx, arg0, 57, 0));
  set(ctx, arg0, 58, get(ctx, arg0, 58, 0));
  set(ctx, arg0, 59, get(ctx, arg0, 59, 0));
  set(ctx, arg0, 60, get(ctx, arg0, 60, 0));
  set(ctx, arg0, 61, get(ctx, arg0, 61, 0));
  set(ctx, arg0, 62, get(ctx, arg0, 62, 0));
  set(ctx, arg0, 63, get(ctx, arg0, 63, 0));
  set(ctx, arg0, 64, get(ctx, arg0, 64, 0));
  set(ctx, arg0, 65, get(ctx, arg0, 65, 0));
  set(ctx, arg0, 66, get(ctx, arg0, 66, 0));
  set(ctx, arg0, 67, get(ctx, arg0, 67, 0));
  set(ctx, arg0, 68, get(ctx, arg0, 68, 0));
  set(ctx, arg0, 69, get(ctx, arg0, 69, 0));
  set(ctx, arg0, 70, get(ctx, arg0, 70, 0));
  set(ctx, arg0, 71, get(ctx, arg0, 71, 0));
  set(ctx, arg0, 72, get(ctx, arg0, 72, 0));
  set(ctx, arg0, 73, get(ctx, arg0, 73, 0));
  set(ctx, arg0, 74, get(ctx, arg0, 74, 0));
  set(ctx, arg0, 75, get(ctx, arg0, 75, 0));
  set(ctx, arg0, 76, get(ctx, arg0, 76, 0));
  set(ctx, arg0, 77, get(ctx, arg0, 77, 0));
  set(ctx, arg0, 78, get(ctx, arg0, 78, 0));
  set(ctx, arg0, 79, get(ctx, arg0, 79, 0));
  set(ctx, arg0, 80, get(ctx, arg0, 80, 0));
  set(ctx, arg0, 81, get(ctx, arg0, 81, 0));
  set(ctx, arg0, 82, get(ctx, arg0, 82, 0));
  set(ctx, arg0, 83, get(ctx, arg0, 83, 0));
  set(ctx, arg0, 84, get(ctx, arg0, 84, 0));
  set(ctx, arg0, 85, get(ctx, arg0, 85, 0));
  set(ctx, arg0, 86, get(ctx, arg0, 86, 0));
  set(ctx, arg0, 87, get(ctx, arg0, 87, 0));
  set(ctx, arg0, 88, get(ctx, arg0, 88, 0));
  set(ctx, arg0, 89, get(ctx, arg0, 89, 0));
  set(ctx, arg0, 90, get(ctx, arg0, 90, 0));
  set(ctx, arg0, 91, get(ctx, arg0, 91, 0));
  set(ctx, arg0, 92, get(ctx, arg0, 92, 0));
  set(ctx, arg0, 93, get(ctx, arg0, 93, 0));
  set(ctx, arg0, 94, get(ctx, arg0, 94, 0));
  set(ctx, arg0, 95, get(ctx, arg0, 95, 0));
  set(ctx, arg0, 96, get(ctx, arg0, 96, 0));
  set(ctx, arg0, 97, get(ctx, arg0, 97, 0));
  set(ctx, arg0, 98, get(ctx, arg0, 98, 0));
  set(ctx, arg0, 99, get(ctx, arg0, 99, 0));
  set(ctx, arg0, 100, get(ctx, arg0, 100, 0));
  set(ctx, arg0, 101, get(ctx, arg0, 101, 0));
  set(ctx, arg0, 102, get(ctx, arg0, 102, 0));
  set(ctx, arg0, 103, get(ctx, arg0, 103, 0));
  set(ctx, arg0, 104, get(ctx, arg0, 104, 0));
  set(ctx, arg0, 105, get(ctx, arg0, 105, 0));
  set(ctx, arg0, 106, get(ctx, arg0, 106, 0));
  set(ctx, arg0, 107, get(ctx, arg0, 107, 0));
  set(ctx, arg0, 108, get(ctx, arg0, 108, 0));
  set(ctx, arg0, 109, get(ctx, arg0, 109, 0));
  set(ctx, arg0, 110, get(ctx, arg0, 110, 0));
  set(ctx, arg0, 111, get(ctx, arg0, 111, 0));
  set(ctx, arg0, 112, get(ctx, arg0, 112, 0));
  set(ctx, arg0, 113, get(ctx, arg0, 113, 0));
  set(ctx, arg0, 114, get(ctx, arg0, 114, 0));
  set(ctx, arg0, 115, get(ctx, arg0, 115, 0));
  set(ctx, arg0, 116, get(ctx, arg0, 116, 0));
  set(ctx, arg0, 117, get(ctx, arg0, 117, 0));
  set(ctx, arg0, 118, get(ctx, arg0, 118, 0));
  set(ctx, arg0, 119, get(ctx, arg0, 119, 0));
  set(ctx, arg0, 120, get(ctx, arg0, 120, 0));
  set(ctx, arg0, 121, get(ctx, arg0, 121, 0));
  set(ctx, arg0, 122, get(ctx, arg0, 122, 0));
  set(ctx, arg0, 123, get(ctx, arg0, 123, 0));
  set(ctx, arg0, 124, get(ctx, arg0, 124, 0));
  set(ctx, arg0, 125, get(ctx, arg0, 125, 0));
  set(ctx, arg0, 126, get(ctx, arg0, 126, 0));
  set(ctx, arg0, 127, get(ctx, arg0, 127, 0));
  set(ctx, arg0, 128, get(ctx, arg0, 128, 0));
  set(ctx, arg0, 129, get(ctx, arg0, 129, 0));
  set(ctx, arg0, 130, get(ctx, arg0, 130, 0));
  set(ctx, arg0, 131, get(ctx, arg0, 131, 0));
  set(ctx, arg0, 132, get(ctx, arg0, 132, 0));
  set(ctx, arg0, 133, get(ctx, arg0, 133, 0));
  set(ctx, arg0, 134, get(ctx, arg0, 134, 0));
  set(ctx, arg0, 135, get(ctx, arg0, 135, 0));
  set(ctx, arg0, 136, get(ctx, arg0, 136, 0));
  set(ctx, arg0, 137, get(ctx, arg0, 137, 0));
  set(ctx, arg0, 138, get(ctx, arg0, 138, 0));
  set(ctx, arg0, 139, get(ctx, arg0, 139, 0));
  set(ctx, arg0, 140, get(ctx, arg0, 140, 0));
  set(ctx, arg0, 141, get(ctx, arg0, 141, 0));
  set(ctx, arg0, 142, get(ctx, arg0, 142, 0));
  set(ctx, arg0, 143, get(ctx, arg0, 143, 0));
  set(ctx, arg0, 144, get(ctx, arg0, 144, 0));
  set(ctx, arg0, 145, get(ctx, arg0, 145, 0));
  set(ctx, arg0, 146, get(ctx, arg0, 146, 0));
  set(ctx, arg0, 147, get(ctx, arg0, 147, 0));
  set(ctx, arg0, 148, get(ctx, arg0, 148, 0));
  set(ctx, arg0, 149, get(ctx, arg0, 149, 0));
  set(ctx, arg0, 150, get(ctx, arg0, 150, 0));
  set(ctx, arg0, 151, get(ctx, arg0, 151, 0));
  set(ctx, arg0, 152, get(ctx, arg0, 152, 0));
  set(ctx, arg0, 153, get(ctx, arg0, 153, 0));
  set(ctx, arg0, 154, get(ctx, arg0, 154, 0));
  set(ctx, arg0, 155, get(ctx, arg0, 155, 0));
  set(ctx, arg0, 156, get(ctx, arg0, 156, 0));
  set(ctx, arg0, 157, get(ctx, arg0, 157, 0));
  set(ctx, arg0, 158, get(ctx, arg0, 158, 0));
  set(ctx, arg0, 159, get(ctx, arg0, 159, 0));
  set(ctx, arg0, 160, get(ctx, arg0, 160, 0));
  set(ctx, arg0, 161, get(ctx, arg0, 161, 0));
  set(ctx, arg0, 162, get(ctx, arg0, 162, 0));
  set(ctx, arg0, 163, get(ctx, arg0, 163, 0));
  set(ctx, arg0, 164, get(ctx, arg0, 164, 0));
  set(ctx, arg0, 165, get(ctx, arg0, 165, 0));
  set(ctx, arg0, 166, get(ctx, arg0, 166, 0));
  set(ctx, arg0, 167, get(ctx, arg0, 167, 0));
  set(ctx, arg0, 168, get(ctx, arg0, 168, 0));
  set(ctx, arg0, 169, get(ctx, arg0, 169, 0));
  set(ctx, arg0, 170, get(ctx, arg0, 170, 0));
  set(ctx, arg0, 171, get(ctx, arg0, 171, 0));
  set(ctx, arg0, 172, get(ctx, arg0, 172, 0));
  set(ctx, arg0, 173, get(ctx, arg0, 173, 0));
  set(ctx, arg0, 174, get(ctx, arg0, 174, 0));
  set(ctx, arg0, 175, get(ctx, arg0, 175, 0));
  set(ctx, arg0, 176, get(ctx, arg0, 176, 0));
  set(ctx, arg0, 177, get(ctx, arg0, 177, 0));
  set(ctx, arg0, 178, get(ctx, arg0, 178, 0));
  set(ctx, arg0, 179, get(ctx, arg0, 179, 0));
  set(ctx, arg0, 180, get(ctx, arg0, 180, 0));
  set(ctx, arg0, 181, get(ctx, arg0, 181, 0));
  set(ctx, arg0, 182, get(ctx, arg0, 182, 0));
  set(ctx, arg0, 183, get(ctx, arg0, 183, 0));
  set(ctx, arg0, 184, get(ctx, arg0, 184, 0));
  set(ctx, arg0, 185, get(ctx, arg0, 185, 0));
  set(ctx, arg0, 186, get(ctx, arg0, 186, 0));
  set(ctx, arg0, 187, get(ctx, arg0, 187, 0));
  set(ctx, arg0, 188, get(ctx, arg0, 188, 0));
  set(ctx, arg0, 189, get(ctx, arg0, 189, 0));
  set(ctx, arg0, 190, get(ctx, arg0, 190, 0));
  set(ctx, arg0, 191, get(ctx, arg0, 191, 0));
  set(ctx, arg0, 192, get(ctx, arg0, 192, 0));
  set(ctx, arg0, 193, get(ctx, arg0, 193, 0));
  set(ctx, arg0, 194, get(ctx, arg0, 194, 0));
  set(ctx, arg0, 195, get(ctx, arg0, 195, 0));
  set(ctx, arg0, 196, get(ctx, arg0, 196, 0));
  set(ctx, arg0, 197, get(ctx, arg0, 197, 0));
  set(ctx, arg0, 198, get(ctx, arg0, 198, 0));
  set(ctx, arg0, 199, get(ctx, arg0, 199, 0));
  set(ctx, arg0, 200, get(ctx, arg0, 200, 0));
  set(ctx, arg0, 201, get(ctx, arg0, 201, 0));
  set(ctx, arg0, 202, get(ctx, arg0, 202, 0));
  set(ctx, arg0, 203, get(ctx, arg0, 203, 0));
  set(ctx, arg0, 204, get(ctx, arg0, 204, 0));
  set(ctx, arg0, 205, get(ctx, arg0, 205, 0));
  set(ctx, arg0, 206, get(ctx, arg0, 206, 0));
  set(ctx, arg0, 207, get(ctx, arg0, 207, 0));
  set(ctx, arg0, 208, get(ctx, arg0, 208, 0));
  set(ctx, arg0, 209, get(ctx, arg0, 209, 0));
  set(ctx, arg0, 210, get(ctx, arg0, 210, 0));
  set(ctx, arg0, 211, get(ctx, arg0, 211, 0));
  set(ctx, arg0, 212, get(ctx, arg0, 212, 0));
  set(ctx, arg0, 213, get(ctx, arg0, 213, 0));
  set(ctx, arg0, 214, get(ctx, arg0, 214, 0));
  set(ctx, arg0, 215, get(ctx, arg0, 215, 0));
  set(ctx, arg0, 216, get(ctx, arg0, 216, 0));
  set(ctx, arg0, 217, get(ctx, arg0, 217, 0));
  set(ctx, arg0, 218, get(ctx, arg0, 218, 0));
  set(ctx, arg0, 219, get(ctx, arg0, 219, 0));
  set(ctx, arg0, 220, get(ctx, arg0, 220, 0));
  set(ctx, arg0, 221, get(ctx, arg0, 221, 0));
  set(ctx, arg0, 222, get(ctx, arg0, 222, 0));
  set(ctx, arg0, 223, get(ctx, arg0, 223, 0));
  set(ctx, arg0, 224, get(ctx, arg0, 224, 0));
  set(ctx, arg0, 225, get(ctx, arg0, 225, 0));
  set(ctx, arg0, 226, get(ctx, arg0, 226, 0));
  set(ctx, arg0, 227, get(ctx, arg0, 227, 0));
  set(ctx, arg0, 228, get(ctx, arg0, 228, 0));
  set(ctx, arg0, 229, get(ctx, arg0, 229, 0));
  set(ctx, arg0, 230, get(ctx, arg0, 230, 0));
  set(ctx, arg0, 231, get(ctx, arg0, 231, 0));
  set(ctx, arg0, 232, get(ctx, arg0, 232, 0));
  set(ctx, arg0, 233, get(ctx, arg0, 233, 0));
  set(ctx, arg0, 234, get(ctx, arg0, 234, 0));
  set(ctx, arg0, 235, get(ctx, arg0, 235, 0));
  set(ctx, arg0, 236, get(ctx, arg0, 236, 0));
  set(ctx, arg0, 237, get(ctx, arg0, 237, 0));
  set(ctx, arg0, 238, get(ctx, arg0, 238, 0));
  set(ctx, arg0, 239, get(ctx, arg0, 239, 0));
  set(ctx, arg0, 240, get(ctx, arg0, 240, 0));
  set(ctx, arg0, 241, get(ctx, arg0, 241, 0));
  set(ctx, arg0, 242, get(ctx, arg0, 242, 0));
  set(ctx, arg0, 243, get(ctx, arg0, 243, 0));
  set(ctx, arg0, 244, get(ctx, arg0, 244, 0));
  set(ctx, arg0, 245, get(ctx, arg0, 245, 0));
  set(ctx, arg0, 246, get(ctx, arg0, 246, 0));
  set(ctx, arg0, 247, get(ctx, arg0, 247, 0));
  set(ctx, arg0, 248, get(ctx, arg0, 248, 0));
  set(ctx, arg0, 249, get(ctx, arg0, 249, 0));
  set(ctx, arg0, 250, get(ctx, arg0, 250, 0));
  set(ctx, arg0, 251, get(ctx, arg0, 251, 0));
  set(ctx, arg0, 252, get(ctx, arg0, 252, 0));
  set(ctx, arg0, 253, get(ctx, arg0, 253, 0));
  set(ctx, arg0, 254, get(ctx, arg0, 254, 0));
  set(ctx, arg0, 255, get(ctx, arg0, 255, 0));
  set(ctx, arg0, 256, get(ctx, arg0, 256, 0));
  set(ctx, arg0, 257, get(ctx, arg0, 257, 0));
  set(ctx, arg0, 258, get(ctx, arg0, 258, 0));
  set(ctx, arg0, 259, get(ctx, arg0, 259, 0));
  set(ctx, arg0, 260, get(ctx, arg0, 260, 0));
  set(ctx, arg0, 261, get(ctx, arg0, 261, 0));
  set(ctx, arg0, 262, get(ctx, arg0, 262, 0));
  set(ctx, arg0, 263, get(ctx, arg0, 263, 0));
  set(ctx, arg0, 264, get(ctx, arg0, 264, 0));
  set(ctx, arg0, 265, get(ctx, arg0, 265, 0));
  set(ctx, arg0, 266, get(ctx, arg0, 266, 0));
  set(ctx, arg0, 267, get(ctx, arg0, 267, 0));
  set(ctx, arg0, 268, get(ctx, arg0, 268, 0));
  set(ctx, arg0, 269, get(ctx, arg0, 269, 0));
  set(ctx, arg0, 270, get(ctx, arg0, 270, 0));
  set(ctx, arg0, 271, get(ctx, arg0, 271, 0));
  set(ctx, arg0, 272, get(ctx, arg0, 272, 0));
  set(ctx, arg0, 273, get(ctx, arg0, 273, 0));
  set(ctx, arg0, 274, get(ctx, arg0, 274, 0));
  set(ctx, arg0, 275, get(ctx, arg0, 275, 0));
  set(ctx, arg0, 276, get(ctx, arg0, 276, 0));
  set(ctx, arg0, 277, get(ctx, arg0, 277, 0));
  set(ctx, arg0, 278, get(ctx, arg0, 278, 0));
  set(ctx, arg0, 279, get(ctx, arg0, 279, 0));
  set(ctx, arg0, 280, get(ctx, arg0, 280, 0));
  set(ctx, arg0, 281, get(ctx, arg0, 281, 0));
  set(ctx, arg0, 282, get(ctx, arg0, 282, 0));
  set(ctx, arg0, 283, get(ctx, arg0, 283, 0));
  set(ctx, arg0, 284, get(ctx, arg0, 284, 0));
  set(ctx, arg0, 285, get(ctx, arg0, 285, 0));
  set(ctx, arg0, 286, get(ctx, arg0, 286, 0));
  set(ctx, arg0, 287, get(ctx, arg0, 287, 0));
  set(ctx, arg0, 288, get(ctx, arg0, 288, 0));
  set(ctx, arg0, 289, get(ctx, arg0, 289, 0));
  set(ctx, arg0, 290, get(ctx, arg0, 290, 0));
  set(ctx, arg0, 291, get(ctx, arg0, 291, 0));
  set(ctx, arg0, 292, get(ctx, arg0, 292, 0));
  set(ctx, arg0, 293, get(ctx, arg0, 293, 0));
  set(ctx, arg0, 294, get(ctx, arg0, 294, 0));
  set(ctx, arg0, 295, get(ctx, arg0, 295, 0));
  set(ctx, arg0, 296, get(ctx, arg0, 296, 0));
  set(ctx, arg0, 297, get(ctx, arg0, 297, 0));
  set(ctx, arg0, 298, get(ctx, arg0, 298, 0));
  set(ctx, arg0, 299, get(ctx, arg0, 299, 0));
  set(ctx, arg0, 300, get(ctx, arg0, 300, 0));
  set(ctx, arg0, 301, get(ctx, arg0, 301, 0));
  set(ctx, arg0, 302, get(ctx, arg0, 302, 0));
  set(ctx, arg0, 303, get(ctx, arg0, 303, 0));
  set(ctx, arg0, 304, get(ctx, arg0, 304, 0));
  set(ctx, arg0, 305, get(ctx, arg0, 305, 0));
  set(ctx, arg0, 306, get(ctx, arg0, 306, 0));
  set(ctx, arg0, 307, get(ctx, arg0, 307, 0));
  set(ctx, arg0, 308, get(ctx, arg0, 308, 0));
  set(ctx, arg0, 309, get(ctx, arg0, 309, 0));
  set(ctx, arg0, 310, get(ctx, arg0, 310, 0));
  set(ctx, arg0, 311, get(ctx, arg0, 311, 0));
  set(ctx, arg0, 312, get(ctx, arg0, 312, 0));
  set(ctx, arg0, 313, get(ctx, arg0, 313, 0));
  set(ctx, arg0, 314, get(ctx, arg0, 314, 0));
  set(ctx, arg0, 315, get(ctx, arg0, 315, 0));
  set(ctx, arg0, 316, get(ctx, arg0, 316, 0));
  set(ctx, arg0, 317, get(ctx, arg0, 317, 0));
  set(ctx, arg0, 318, get(ctx, arg0, 318, 0));
  set(ctx, arg0, 319, get(ctx, arg0, 319, 0));
  set(ctx, arg0, 320, get(ctx, arg0, 320, 0));
  set(ctx, arg0, 321, get(ctx, arg0, 321, 0));
  set(ctx, arg0, 322, get(ctx, arg0, 322, 0));
  set(ctx, arg0, 323, get(ctx, arg0, 323, 0));
  set(ctx, arg0, 324, get(ctx, arg0, 324, 0));
  set(ctx, arg0, 325, get(ctx, arg0, 325, 0));
  set(ctx, arg0, 326, get(ctx, arg0, 326, 0));
  set(ctx, arg0, 327, get(ctx, arg0, 327, 0));
  set(ctx, arg0, 328, get(ctx, arg0, 328, 0));
  set(ctx, arg0, 329, get(ctx, arg0, 329, 0));
  set(ctx, arg0, 330, get(ctx, arg0, 330, 0));
  set(ctx, arg0, 331, get(ctx, arg0, 331, 0));
  set(ctx, arg0, 332, get(ctx, arg0, 332, 0));
  set(ctx, arg0, 333, get(ctx, arg0, 333, 0));
  set(ctx, arg0, 334, get(ctx, arg0, 334, 0));
  set(ctx, arg0, 335, get(ctx, arg0, 335, 0));
  set(ctx, arg0, 336, get(ctx, arg0, 336, 0));
  set(ctx, arg0, 337, get(ctx, arg0, 337, 0));
  set(ctx, arg0, 338, get(ctx, arg0, 338, 0));
  set(ctx, arg0, 339, get(ctx, arg0, 339, 0));
  set(ctx, arg0, 340, get(ctx, arg0, 340, 0));
  set(ctx, arg0, 341, get(ctx, arg0, 341, 0));
  set(ctx, arg0, 342, get(ctx, arg0, 342, 0));
  set(ctx, arg0, 343, get(ctx, arg0, 343, 0));
  set(ctx, arg0, 344, get(ctx, arg0, 344, 0));
  set(ctx, arg0, 345, get(ctx, arg0, 345, 0));
  set(ctx, arg0, 346, get(ctx, arg0, 346, 0));
  set(ctx, arg0, 347, get(ctx, arg0, 347, 0));
  set(ctx, arg0, 348, get(ctx, arg0, 348, 0));
  set(ctx, arg0, 349, get(ctx, arg0, 349, 0));
  set(ctx, arg0, 350, get(ctx, arg0, 350, 0));
  set(ctx, arg0, 351, get(ctx, arg0, 351, 0));
  set(ctx, arg0, 352, get(ctx, arg0, 352, 0));
  set(ctx, arg0, 353, get(ctx, arg0, 353, 0));
  set(ctx, arg0, 354, get(ctx, arg0, 354, 0));
  set(ctx, arg0, 355, get(ctx, arg0, 355, 0));
  set(ctx, arg0, 356, get(ctx, arg0, 356, 0));
  set(ctx, arg0, 357, get(ctx, arg0, 357, 0));
  set(ctx, arg0, 358, get(ctx, arg0, 358, 0));
  set(ctx, arg0, 359, get(ctx, arg0, 359, 0));
  set(ctx, arg0, 360, get(ctx, arg0, 360, 0));
  set(ctx, arg0, 361, get(ctx, arg0, 361, 0));
  set(ctx, arg0, 362, get(ctx, arg0, 362, 0));
  set(ctx, arg0, 363, get(ctx, arg0, 363, 0));
  set(ctx, arg0, 364, get(ctx, arg0, 364, 0));
  set(ctx, arg0, 365, get(ctx, arg0, 365, 0));
  set(ctx, arg0, 366, get(ctx, arg0, 366, 0));
  set(ctx, arg0, 367, get(ctx, arg0, 367, 0));
  set(ctx, arg0, 368, get(ctx, arg0, 368, 0));
  set(ctx, arg0, 369, get(ctx, arg0, 369, 0));
  set(ctx, arg0, 370, get(ctx, arg0, 370, 0));
  set(ctx, arg0, 371, get(ctx, arg0, 371, 0));
  set(ctx, arg0, 372, get(ctx, arg0, 372, 0));
  set(ctx, arg0, 373, get(ctx, arg0, 373, 0));
  set(ctx, arg0, 374, get(ctx, arg0, 374, 0));
  set(ctx, arg0, 375, get(ctx, arg0, 375, 0));
  set(ctx, arg0, 376, get(ctx, arg0, 376, 0));
  set(ctx, arg0, 377, get(ctx, arg0, 377, 0));
  set(ctx, arg0, 378, get(ctx, arg0, 378, 0));
  set(ctx, arg0, 379, get(ctx, arg0, 379, 0));
  set(ctx, arg0, 380, get(ctx, arg0, 380, 0));
  set(ctx, arg0, 381, get(ctx, arg0, 381, 0));
  set(ctx, arg0, 382, get(ctx, arg0, 382, 0));
  set(ctx, arg0, 383, get(ctx, arg0, 383, 0));
  set(ctx, arg0, 384, get(ctx, arg0, 384, 0));
  set(ctx, arg0, 385, get(ctx, arg0, 385, 0));
  set(ctx, arg0, 386, get(ctx, arg0, 386, 0));
  set(ctx, arg0, 387, get(ctx, arg0, 387, 0));
  set(ctx, arg0, 388, get(ctx, arg0, 388, 0));
  set(ctx, arg0, 389, get(ctx, arg0, 389, 0));
  set(ctx, arg0, 390, get(ctx, arg0, 390, 0));
  set(ctx, arg0, 391, get(ctx, arg0, 391, 0));
  set(ctx, arg0, 392, get(ctx, arg0, 392, 0));
  set(ctx, arg0, 393, get(ctx, arg0, 393, 0));
  set(ctx, arg0, 394, get(ctx, arg0, 394, 0));
  set(ctx, arg0, 395, get(ctx, arg0, 395, 0));
  set(ctx, arg0, 396, get(ctx, arg0, 396, 0));
  set(ctx, arg0, 397, get(ctx, arg0, 397, 0));
  set(ctx, arg0, 398, get(ctx, arg0, 398, 0));
  set(ctx, arg0, 399, get(ctx, arg0, 399, 0));
  set(ctx, arg0, 400, get(ctx, arg0, 400, 0));
  set(ctx, arg0, 401, get(ctx, arg0, 401, 0));
  set(ctx, arg0, 402, get(ctx, arg0, 402, 0));
  set(ctx, arg0, 403, get(ctx, arg0, 403, 0));
  set(ctx, arg0, 404, get(ctx, arg0, 404, 0));
  set(ctx, arg0, 405, get(ctx, arg0, 405, 0));
  set(ctx, arg0, 406, get(ctx, arg0, 406, 0));
  set(ctx, arg0, 407, get(ctx, arg0, 407, 0));
  set(ctx, arg0, 408, get(ctx, arg0, 408, 0));
  set(ctx, arg0, 409, get(ctx, arg0, 409, 0));
  set(ctx, arg0, 410, get(ctx, arg0, 410, 0));
  set(ctx, arg0, 411, get(ctx, arg0, 411, 0));
  set(ctx, arg0, 412, get(ctx, arg0, 412, 0));
  set(ctx, arg0, 413, get(ctx, arg0, 413, 0));
  set(ctx, arg0, 414, get(ctx, arg0, 414, 0));
  set(ctx, arg0, 415, get(ctx, arg0, 415, 0));
  set(ctx, arg0, 416, get(ctx, arg0, 416, 0));
  set(ctx, arg0, 417, get(ctx, arg0, 417, 0));
  set(ctx, arg0, 418, get(ctx, arg0, 418, 0));
  set(ctx, arg0, 419, get(ctx, arg0, 419, 0));
  set(ctx, arg0, 420, get(ctx, arg0, 420, 0));
  set(ctx, arg0, 421, get(ctx, arg0, 421, 0));
  set(ctx, arg0, 422, get(ctx, arg0, 422, 0));
  set(ctx, arg0, 423, get(ctx, arg0, 423, 0));
  set(ctx, arg0, 424, get(ctx, arg0, 424, 0));
  set(ctx, arg0, 425, get(ctx, arg0, 425, 0));
  set(ctx, arg0, 426, get(ctx, arg0, 426, 0));
  set(ctx, arg0, 427, get(ctx, arg0, 427, 0));
  set(ctx, arg0, 428, get(ctx, arg0, 428, 0));
  set(ctx, arg0, 429, get(ctx, arg0, 429, 0));
  set(ctx, arg0, 430, get(ctx, arg0, 430, 0));
  set(ctx, arg0, 431, get(ctx, arg0, 431, 0));
  set(ctx, arg0, 432, get(ctx, arg0, 432, 0));
  set(ctx, arg0, 433, get(ctx, arg0, 433, 0));
  set(ctx, arg0, 434, get(ctx, arg0, 434, 0));
  set(ctx, arg0, 435, get(ctx, arg0, 435, 0));
  set(ctx, arg0, 436, get(ctx, arg0, 436, 0));
  set(ctx, arg0, 437, get(ctx, arg0, 437, 0));
  set(ctx, arg0, 438, get(ctx, arg0, 438, 0));
  set(ctx, arg0, 439, get(ctx, arg0, 439, 0));
  set(ctx, arg0, 440, get(ctx, arg0, 440, 0));
  set(ctx, arg0, 441, get(ctx, arg0, 441, 0));
  set(ctx, arg0, 442, get(ctx, arg0, 442, 0));
  set(ctx, arg0, 443, get(ctx, arg0, 443, 0));
  set(ctx, arg0, 444, get(ctx, arg0, 444, 0));
  set(ctx, arg0, 445, get(ctx, arg0, 445, 0));
  set(ctx, arg0, 446, get(ctx, arg0, 446, 0));
  set(ctx, arg0, 447, get(ctx, arg0, 447, 0));
  set(ctx, arg0, 448, get(ctx, arg0, 448, 0));
  set(ctx, arg0, 449, get(ctx, arg0, 449, 0));
  set(ctx, arg0, 450, get(ctx, arg0, 450, 0));
  set(ctx, arg0, 451, get(ctx, arg0, 451, 0));
  set(ctx, arg0, 452, get(ctx, arg0, 452, 0));
  set(ctx, arg0, 453, get(ctx, arg0, 453, 0));
  set(ctx, arg0, 454, get(ctx, arg0, 454, 0));
  set(ctx, arg0, 455, get(ctx, arg0, 455, 0));
  set(ctx, arg0, 456, get(ctx, arg0, 456, 0));
  set(ctx, arg0, 457, get(ctx, arg0, 457, 0));
  set(ctx, arg0, 458, get(ctx, arg0, 458, 0));
  set(ctx, arg0, 459, get(ctx, arg0, 459, 0));
  set(ctx, arg0, 460, get(ctx, arg0, 460, 0));
  set(ctx, arg0, 461, get(ctx, arg0, 461, 0));
  set(ctx, arg0, 462, get(ctx, arg0, 462, 0));
  set(ctx, arg0, 463, get(ctx, arg0, 463, 0));
  set(ctx, arg0, 464, get(ctx, arg0, 464, 0));
  set(ctx, arg0, 465, get(ctx, arg0, 465, 0));
  set(ctx, arg0, 466, get(ctx, arg0, 466, 0));
  set(ctx, arg0, 467, get(ctx, arg0, 467, 0));
  set(ctx, arg0, 468, get(ctx, arg0, 468, 0));
  set(ctx, arg0, 469, get(ctx, arg0, 469, 0));
  set(ctx, arg0, 470, get(ctx, arg0, 470, 0));
  set(ctx, arg0, 471, get(ctx, arg0, 471, 0));
  set(ctx, arg0, 472, get(ctx, arg0, 472, 0));
  set(ctx, arg0, 473, get(ctx, arg0, 473, 0));
  set(ctx, arg0, 474, get(ctx, arg0, 474, 0));
  set(ctx, arg0, 475, get(ctx, arg0, 475, 0));
  set(ctx, arg0, 476, get(ctx, arg0, 476, 0));
  set(ctx, arg0, 477, get(ctx, arg0, 477, 0));
  set(ctx, arg0, 478, get(ctx, arg0, 478, 0));
  set(ctx, arg0, 479, get(ctx, arg0, 479, 0));
  set(ctx, arg0, 480, get(ctx, arg0, 480, 0));
  set(ctx, arg0, 481, get(ctx, arg0, 481, 0));
  set(ctx, arg0, 482, get(ctx, arg0, 482, 0));
  set(ctx, arg0, 483, get(ctx, arg0, 483, 0));
  set(ctx, arg0, 484, get(ctx, arg0, 484, 0));
  set(ctx, arg0, 485, get(ctx, arg0, 485, 0));
  set(ctx, arg0, 486, get(ctx, arg0, 486, 0));
  set(ctx, arg0, 487, get(ctx, arg0, 487, 0));
  set(ctx, arg0, 488, get(ctx, arg0, 488, 0));
  set(ctx, arg0, 489, get(ctx, arg0, 489, 0));
  set(ctx, arg0, 490, get(ctx, arg0, 490, 0));
  set(ctx, arg0, 491, get(ctx, arg0, 491, 0));
  set(ctx, arg0, 492, get(ctx, arg0, 492, 0));
  set(ctx, arg0, 493, get(ctx, arg0, 493, 0));
  set(ctx, arg0, 494, get(ctx, arg0, 494, 0));
  set(ctx, arg0, 495, get(ctx, arg0, 495, 0));
  set(ctx, arg0, 496, get(ctx, arg0, 496, 0));
  set(ctx, arg0, 497, get(ctx, arg0, 497, 0));
  set(ctx, arg0, 498, get(ctx, arg0, 498, 0));
  set(ctx, arg0, 499, get(ctx, arg0, 499, 0));
  set(ctx, arg0, 500, get(ctx, arg0, 500, 0));
  set(ctx, arg0, 501, get(ctx, arg0, 501, 0));
  set(ctx, arg0, 502, get(ctx, arg0, 502, 0));
  set(ctx, arg0, 503, get(ctx, arg0, 503, 0));
  set(ctx, arg0, 504, get(ctx, arg0, 504, 0));
  set(ctx, arg0, 505, get(ctx, arg0, 505, 0));
  set(ctx, arg0, 506, get(ctx, arg0, 506, 0));
  set(ctx, arg0, 507, get(ctx, arg0, 507, 0));
  set(ctx, arg0, 508, get(ctx, arg0, 508, 0));
  set(ctx, arg0, 509, get(ctx, arg0, 509, 0));
  set(ctx, arg0, 510, get(ctx, arg0, 510, 0));
  set(ctx, arg0, 511, get(ctx, arg0, 511, 0));
  set(ctx, arg0, 512, get(ctx, arg0, 512, 0));
  set(ctx, arg0, 513, get(ctx, arg0, 513, 0));
  set(ctx, arg0, 514, get(ctx, arg0, 514, 0));
  set(ctx, arg0, 515, get(ctx, arg0, 515, 0));
  set(ctx, arg0, 516, get(ctx, arg0, 516, 0));
  set(ctx, arg0, 517, get(ctx, arg0, 517, 0));
  set(ctx, arg0, 518, get(ctx, arg0, 518, 0));
  set(ctx, arg0, 519, get(ctx, arg0, 519, 0));
  set(ctx, arg0, 520, get(ctx, arg0, 520, 0));
  set(ctx, arg0, 521, get(ctx, arg0, 521, 0));
  set(ctx, arg0, 522, get(ctx, arg0, 522, 0));
  set(ctx, arg0, 523, get(ctx, arg0, 523, 0));
  set(ctx, arg0, 524, get(ctx, arg0, 524, 0));
  set(ctx, arg0, 525, get(ctx, arg0, 525, 0));
  set(ctx, arg0, 526, get(ctx, arg0, 526, 0));
  set(ctx, arg0, 527, get(ctx, arg0, 527, 0));
  set(ctx, arg0, 528, get(ctx, arg0, 528, 0));
  set(ctx, arg0, 529, get(ctx, arg0, 529, 0));
  set(ctx, arg0, 530, get(ctx, arg0, 530, 0));
  set(ctx, arg0, 531, get(ctx, arg0, 531, 0));
  set(ctx, arg0, 532, get(ctx, arg0, 532, 0));
  set(ctx, arg0, 533, get(ctx, arg0, 533, 0));
  set(ctx, arg0, 534, get(ctx, arg0, 534, 0));
  set(ctx, arg0, 535, get(ctx, arg0, 535, 0));
  set(ctx, arg0, 536, get(ctx, arg0, 536, 0));
  set(ctx, arg0, 537, get(ctx, arg0, 537, 0));
  set(ctx, arg0, 538, get(ctx, arg0, 538, 0));
  set(ctx, arg0, 539, get(ctx, arg0, 539, 0));
  set(ctx, arg0, 540, get(ctx, arg0, 540, 0));
  set(ctx, arg0, 541, get(ctx, arg0, 541, 0));
  set(ctx, arg0, 542, get(ctx, arg0, 542, 0));
  set(ctx, arg0, 543, get(ctx, arg0, 543, 0));
  set(ctx, arg0, 544, get(ctx, arg0, 544, 0));
  set(ctx, arg0, 545, get(ctx, arg0, 545, 0));
  set(ctx, arg0, 546, get(ctx, arg0, 546, 0));
  set(ctx, arg0, 547, get(ctx, arg0, 547, 0));
  set(ctx, arg0, 548, get(ctx, arg0, 548, 0));
  set(ctx, arg0, 549, get(ctx, arg0, 549, 0));
  set(ctx, arg0, 550, get(ctx, arg0, 550, 0));
  set(ctx, arg0, 551, get(ctx, arg0, 551, 0));
  set(ctx, arg0, 552, get(ctx, arg0, 552, 0));
  set(ctx, arg0, 553, get(ctx, arg0, 553, 0));
  set(ctx, arg0, 554, get(ctx, arg0, 554, 0));
  set(ctx, arg0, 555, get(ctx, arg0, 555, 0));
  set(ctx, arg0, 556, get(ctx, arg0, 556, 0));
  set(ctx, arg0, 557, get(ctx, arg0, 557, 0));
  set(ctx, arg0, 558, get(ctx, arg0, 558, 0));
  set(ctx, arg0, 559, get(ctx, arg0, 559, 0));
  set(ctx, arg0, 560, get(ctx, arg0, 560, 0));
  set(ctx, arg0, 561, get(ctx, arg0, 561, 0));
  set(ctx, arg0, 562, get(ctx, arg0, 562, 0));
  set(ctx, arg0, 563, get(ctx, arg0, 563, 0));
  set(ctx, arg0, 564, get(ctx, arg0, 564, 0));
  set(ctx, arg0, 565, get(ctx, arg0, 565, 0));
  set(ctx, arg0, 566, get(ctx, arg0, 566, 0));
  set(ctx, arg0, 567, get(ctx, arg0, 567, 0));
  set(ctx, arg0, 568, get(ctx, arg0, 568, 0));
  set(ctx, arg0, 569, get(ctx, arg0, 569, 0));
  set(ctx, arg0, 570, get(ctx, arg0, 570, 0));
  set(ctx, arg0, 571, get(ctx, arg0, 571, 0));
  set(ctx, arg0, 572, get(ctx, arg0, 572, 0));
  set(ctx, arg0, 573, get(ctx, arg0, 573, 0));
  set(ctx, arg0, 574, get(ctx, arg0, 574, 0));
  set(ctx, arg0, 575, get(ctx, arg0, 575, 0));
  set(ctx, arg0, 576, get(ctx, arg0, 576, 0));
  set(ctx, arg0, 577, get(ctx, arg0, 577, 0));
  set(ctx, arg0, 578, get(ctx, arg0, 578, 0));
  set(ctx, arg0, 579, get(ctx, arg0, 579, 0));
  set(ctx, arg0, 580, get(ctx, arg0, 580, 0));
  set(ctx, arg0, 581, get(ctx, arg0, 581, 0));
  set(ctx, arg0, 582, get(ctx, arg0, 582, 0));
  set(ctx, arg0, 583, get(ctx, arg0, 583, 0));
  set(ctx, arg0, 584, get(ctx, arg0, 584, 0));
  set(ctx, arg0, 585, get(ctx, arg0, 585, 0));
  set(ctx, arg0, 586, get(ctx, arg0, 586, 0));
  set(ctx, arg0, 587, get(ctx, arg0, 587, 0));
  set(ctx, arg0, 588, get(ctx, arg0, 588, 0));
  set(ctx, arg0, 589, get(ctx, arg0, 589, 0));
  set(ctx, arg0, 590, get(ctx, arg0, 590, 0));
  set(ctx, arg0, 591, get(ctx, arg0, 591, 0));
  set(ctx, arg0, 592, get(ctx, arg0, 592, 0));
  set(ctx, arg0, 593, get(ctx, arg0, 593, 0));
  set(ctx, arg0, 594, get(ctx, arg0, 594, 0));
  set(ctx, arg0, 595, get(ctx, arg0, 595, 0));
  set(ctx, arg0, 596, get(ctx, arg0, 596, 0));
  set(ctx, arg0, 597, get(ctx, arg0, 597, 0));
  set(ctx, arg0, 598, get(ctx, arg0, 598, 0));
  set(ctx, arg0, 599, get(ctx, arg0, 599, 0));
  set(ctx, arg0, 600, get(ctx, arg0, 600, 0));
  set(ctx, arg0, 601, get(ctx, arg0, 601, 0));
  set(ctx, arg0, 602, get(ctx, arg0, 602, 0));
  set(ctx, arg0, 603, get(ctx, arg0, 603, 0));
  set(ctx, arg0, 604, get(ctx, arg0, 604, 0));
  set(ctx, arg0, 605, get(ctx, arg0, 605, 0));
  set(ctx, arg0, 606, get(ctx, arg0, 606, 0));
  set(ctx, arg0, 607, get(ctx, arg0, 607, 0));
  set(ctx, arg0, 608, get(ctx, arg0, 608, 0));
  set(ctx, arg0, 609, get(ctx, arg0, 609, 0));
  set(ctx, arg0, 610, get(ctx, arg0, 610, 0));
  set(ctx, arg0, 611, get(ctx, arg0, 611, 0));
  set(ctx, arg0, 612, get(ctx, arg0, 612, 0));
  set(ctx, arg0, 613, get(ctx, arg0, 613, 0));
  set(ctx, arg0, 614, get(ctx, arg0, 614, 0));
  set(ctx, arg0, 615, get(ctx, arg0, 615, 0));
  set(ctx, arg0, 616, get(ctx, arg0, 616, 0));
  set(ctx, arg0, 617, get(ctx, arg0, 617, 0));
  set(ctx, arg0, 618, get(ctx, arg0, 618, 0));
  set(ctx, arg0, 619, get(ctx, arg0, 619, 0));
  set(ctx, arg0, 620, get(ctx, arg0, 620, 0));
  set(ctx, arg0, 621, get(ctx, arg0, 621, 0));
  set(ctx, arg0, 622, get(ctx, arg0, 622, 0));
  set(ctx, arg0, 623, get(ctx, arg0, 623, 0));
  set(ctx, arg0, 624, get(ctx, arg0, 624, 0));
  set(ctx, arg0, 625, get(ctx, arg0, 625, 0));
  set(ctx, arg0, 626, get(ctx, arg0, 626, 0));
  set(ctx, arg0, 627, get(ctx, arg0, 627, 0));
  set(ctx, arg0, 628, get(ctx, arg0, 628, 0));
  set(ctx, arg0, 629, get(ctx, arg0, 629, 0));
  set(ctx, arg0, 630, get(ctx, arg0, 630, 0));
  set(ctx, arg0, 631, get(ctx, arg0, 631, 0));
  set(ctx, arg0, 632, get(ctx, arg0, 632, 0));
  set(ctx, arg0, 633, get(ctx, arg0, 633, 0));
  set(ctx, arg0, 634, get(ctx, arg0, 634, 0));
  set(ctx, arg0, 635, get(ctx, arg0, 635, 0));
  set(ctx, arg0, 636, get(ctx, arg0, 636, 0));
  set(ctx, arg0, 637, get(ctx, arg0, 637, 0));
  set(ctx, arg0, 638, get(ctx, arg0, 638, 0));
  set(ctx, arg0, 639, get(ctx, arg0, 639, 0));
  set(ctx, arg0, 640, get(ctx, arg0, 640, 0));
  set(ctx, arg0, 641, get(ctx, arg0, 641, 0));
  set(ctx, arg0, 642, get(ctx, arg0, 642, 0));
  set(ctx, arg0, 643, get(ctx, arg0, 643, 0));
  set(ctx, arg0, 644, get(ctx, arg0, 644, 0));
  set(ctx, arg0, 645, get(ctx, arg0, 645, 0));
  set(ctx, arg0, 646, get(ctx, arg0, 646, 0));
  set(ctx, arg0, 647, get(ctx, arg0, 647, 0));
  set(ctx, arg0, 648, get(ctx, arg0, 648, 0));
  set(ctx, arg0, 649, get(ctx, arg0, 649, 0));
  set(ctx, arg0, 650, get(ctx, arg0, 650, 0));
  set(ctx, arg0, 651, get(ctx, arg0, 651, 0));
  set(ctx, arg0, 652, get(ctx, arg0, 652, 0));
  set(ctx, arg0, 653, get(ctx, arg0, 653, 0));
  set(ctx, arg0, 654, get(ctx, arg0, 654, 0));
  set(ctx, arg0, 655, get(ctx, arg0, 655, 0));
  set(ctx, arg0, 656, get(ctx, arg0, 656, 0));
  set(ctx, arg0, 657, get(ctx, arg0, 657, 0));
  set(ctx, arg0, 658, get(ctx, arg0, 658, 0));
  set(ctx, arg0, 659, get(ctx, arg0, 659, 0));
  set(ctx, arg0, 660, get(ctx, arg0, 660, 0));
  set(ctx, arg0, 661, get(ctx, arg0, 661, 0));
  set(ctx, arg0, 662, get(ctx, arg0, 662, 0));
  set(ctx, arg0, 663, get(ctx, arg0, 663, 0));
  set(ctx, arg0, 664, get(ctx, arg0, 664, 0));
  set(ctx, arg0, 665, get(ctx, arg0, 665, 0));
  set(ctx, arg0, 666, get(ctx, arg0, 666, 0));
  set(ctx, arg0, 667, get(ctx, arg0, 667, 0));
  set(ctx, arg0, 668, get(ctx, arg0, 668, 0));
  set(ctx, arg0, 669, get(ctx, arg0, 669, 0));
  set(ctx, arg0, 670, get(ctx, arg0, 670, 0));
  set(ctx, arg0, 671, get(ctx, arg0, 671, 0));
  set(ctx, arg0, 672, get(ctx, arg0, 672, 0));
  set(ctx, arg0, 673, get(ctx, arg0, 673, 0));
  set(ctx, arg0, 674, get(ctx, arg0, 674, 0));
  set(ctx, arg0, 675, get(ctx, arg0, 675, 0));
  set(ctx, arg0, 676, get(ctx, arg0, 676, 0));
  set(ctx, arg0, 677, get(ctx, arg0, 677, 0));
  set(ctx, arg0, 678, get(ctx, arg0, 678, 0));
  set(ctx, arg0, 679, get(ctx, arg0, 679, 0));
  set(ctx, arg0, 680, get(ctx, arg0, 680, 0));
  set(ctx, arg0, 681, get(ctx, arg0, 681, 0));
  set(ctx, arg0, 682, get(ctx, arg0, 682, 0));
  set(ctx, arg0, 683, get(ctx, arg0, 683, 0));
  set(ctx, arg0, 684, get(ctx, arg0, 684, 0));
  set(ctx, arg0, 685, get(ctx, arg0, 685, 0));
  set(ctx, arg0, 686, get(ctx, arg0, 686, 0));
  set(ctx, arg0, 687, get(ctx, arg0, 687, 0));
  set(ctx, arg0, 688, get(ctx, arg0, 688, 0));
  set(ctx, arg0, 689, get(ctx, arg0, 689, 0));
  set(ctx, arg0, 690, get(ctx, arg0, 690, 0));
  set(ctx, arg0, 691, get(ctx, arg0, 691, 0));
  set(ctx, arg0, 692, get(ctx, arg0, 692, 0));
  set(ctx, arg0, 693, get(ctx, arg0, 693, 0));
  set(ctx, arg0, 694, get(ctx, arg0, 694, 0));
  set(ctx, arg0, 695, get(ctx, arg0, 695, 0));
  set(ctx, arg0, 696, get(ctx, arg0, 696, 0));
  set(ctx, arg0, 697, get(ctx, arg0, 697, 0));
  set(ctx, arg0, 698, get(ctx, arg0, 698, 0));
  set(ctx, arg0, 699, get(ctx, arg0, 699, 0));
  set(ctx, arg0, 700, get(ctx, arg0, 700, 0));
  set(ctx, arg0, 701, get(ctx, arg0, 701, 0));
  set(ctx, arg0, 702, get(ctx, arg0, 702, 0));
  set(ctx, arg0, 703, get(ctx, arg0, 703, 0));
  set(ctx, arg0, 704, get(ctx, arg0, 704, 0));
  set(ctx, arg0, 705, get(ctx, arg0, 705, 0));
  set(ctx, arg0, 706, get(ctx, arg0, 706, 0));
  set(ctx, arg0, 707, get(ctx, arg0, 707, 0));
  set(ctx, arg0, 708, get(ctx, arg0, 708, 0));
  set(ctx, arg0, 709, get(ctx, arg0, 709, 0));
  set(ctx, arg0, 710, get(ctx, arg0, 710, 0));
  set(ctx, arg0, 711, get(ctx, arg0, 711, 0));
  set(ctx, arg0, 712, get(ctx, arg0, 712, 0));
  set(ctx, arg0, 713, get(ctx, arg0, 713, 0));
  set(ctx, arg0, 714, get(ctx, arg0, 714, 0));
  set(ctx, arg0, 715, get(ctx, arg0, 715, 0));
  set(ctx, arg0, 716, get(ctx, arg0, 716, 0));
  set(ctx, arg0, 717, get(ctx, arg0, 717, 0));
  set(ctx, arg0, 718, get(ctx, arg0, 718, 0));
  set(ctx, arg0, 719, get(ctx, arg0, 719, 0));
  set(ctx, arg0, 720, get(ctx, arg0, 720, 0));
  set(ctx, arg0, 721, get(ctx, arg0, 721, 0));
  set(ctx, arg0, 722, get(ctx, arg0, 722, 0));
  set(ctx, arg0, 723, get(ctx, arg0, 723, 0));
  set(ctx, arg0, 724, get(ctx, arg0, 724, 0));
  set(ctx, arg0, 725, get(ctx, arg0, 725, 0));
  set(ctx, arg0, 726, get(ctx, arg0, 726, 0));
  set(ctx, arg0, 727, get(ctx, arg0, 727, 0));
  set(ctx, arg0, 728, get(ctx, arg0, 728, 0));
  set(ctx, arg0, 729, get(ctx, arg0, 729, 0));
  set(ctx, arg0, 730, get(ctx, arg0, 730, 0));
  set(ctx, arg0, 731, get(ctx, arg0, 731, 0));
  set(ctx, arg0, 732, get(ctx, arg0, 732, 0));
  set(ctx, arg0, 733, get(ctx, arg0, 733, 0));
  set(ctx, arg0, 734, get(ctx, arg0, 734, 0));
  set(ctx, arg0, 735, get(ctx, arg0, 735, 0));
  set(ctx, arg0, 736, get(ctx, arg0, 736, 0));
  set(ctx, arg0, 737, get(ctx, arg0, 737, 0));
  set(ctx, arg0, 738, get(ctx, arg0, 738, 0));
  set(ctx, arg0, 739, get(ctx, arg0, 739, 0));
  set(ctx, arg0, 740, get(ctx, arg0, 740, 0));
  set(ctx, arg0, 741, get(ctx, arg0, 741, 0));
  set(ctx, arg0, 742, get(ctx, arg0, 742, 0));
  set(ctx, arg0, 743, get(ctx, arg0, 743, 0));
  set(ctx, arg0, 744, get(ctx, arg0, 744, 0));
  set(ctx, arg0, 745, get(ctx, arg0, 745, 0));
  set(ctx, arg0, 746, get(ctx, arg0, 746, 0));
  set(ctx, arg0, 747, get(ctx, arg0, 747, 0));
  set(ctx, arg0, 748, get(ctx, arg0, 748, 0));
  set(ctx, arg0, 749, get(ctx, arg0, 749, 0));
  set(ctx, arg0, 750, get(ctx, arg0, 750, 0));
  set(ctx, arg0, 751, get(ctx, arg0, 751, 0));
  set(ctx, arg0, 752, get(ctx, arg0, 752, 0));
  set(ctx, arg0, 753, get(ctx, arg0, 753, 0));
  set(ctx, arg0, 754, get(ctx, arg0, 754, 0));
  set(ctx, arg0, 755, get(ctx, arg0, 755, 0));
  set(ctx, arg0, 756, get(ctx, arg0, 756, 0));
  set(ctx, arg0, 757, get(ctx, arg0, 757, 0));
  set(ctx, arg0, 758, get(ctx, arg0, 758, 0));
  set(ctx, arg0, 759, get(ctx, arg0, 759, 0));
  set(ctx, arg0, 760, get(ctx, arg0, 760, 0));
  set(ctx, arg0, 761, get(ctx, arg0, 761, 0));
  set(ctx, arg0, 762, get(ctx, arg0, 762, 0));
  set(ctx, arg0, 763, get(ctx, arg0, 763, 0));
  set(ctx, arg0, 764, get(ctx, arg0, 764, 0));
  set(ctx, arg0, 765, get(ctx, arg0, 765, 0));
  set(ctx, arg0, 766, get(ctx, arg0, 766, 0));
  set(ctx, arg0, 767, get(ctx, arg0, 767, 0));
  set(ctx, arg0, 768, get(ctx, arg0, 768, 0));
  set(ctx, arg0, 769, get(ctx, arg0, 769, 0));
  set(ctx, arg0, 770, get(ctx, arg0, 770, 0));
  set(ctx, arg0, 771, get(ctx, arg0, 771, 0));
  set(ctx, arg0, 772, get(ctx, arg0, 772, 0));
  set(ctx, arg0, 773, get(ctx, arg0, 773, 0));
  set(ctx, arg0, 774, get(ctx, arg0, 774, 0));
  set(ctx, arg0, 775, get(ctx, arg0, 775, 0));
  set(ctx, arg0, 776, get(ctx, arg0, 776, 0));
  set(ctx, arg0, 777, get(ctx, arg0, 777, 0));
  set(ctx, arg0, 778, get(ctx, arg0, 778, 0));
  set(ctx, arg0, 779, get(ctx, arg0, 779, 0));
  set(ctx, arg0, 780, get(ctx, arg0, 780, 0));
  set(ctx, arg0, 781, get(ctx, arg0, 781, 0));
  set(ctx, arg0, 782, get(ctx, arg0, 782, 0));
  set(ctx, arg0, 783, get(ctx, arg0, 783, 0));
  set(ctx, arg0, 784, get(ctx, arg0, 784, 0));
  set(ctx, arg0, 785, get(ctx, arg0, 785, 0));
  set(ctx, arg0, 786, get(ctx, arg0, 786, 0));
  set(ctx, arg0, 787, get(ctx, arg0, 787, 0));
  set(ctx, arg0, 788, get(ctx, arg0, 788, 0));
  set(ctx, arg0, 789, get(ctx, arg0, 789, 0));
  set(ctx, arg0, 790, get(ctx, arg0, 790, 0));
  set(ctx, arg0, 791, get(ctx, arg0, 791, 0));
  set(ctx, arg0, 792, get(ctx, arg0, 792, 0));
  set(ctx, arg0, 793, get(ctx, arg0, 793, 0));
  set(ctx, arg0, 794, get(ctx, arg0, 794, 0));
  set(ctx, arg0, 795, get(ctx, arg0, 795, 0));
  set(ctx, arg0, 796, get(ctx, arg0, 796, 0));
  set(ctx, arg0, 797, get(ctx, arg0, 797, 0));
  set(ctx, arg0, 798, get(ctx, arg0, 798, 0));
  set(ctx, arg0, 799, get(ctx, arg0, 799, 0));
  set(ctx, arg0, 800, get(ctx, arg0, 800, 0));
  set(ctx, arg0, 801, get(ctx, arg0, 801, 0));
  set(ctx, arg0, 802, get(ctx, arg0, 802, 0));
  set(ctx, arg0, 803, get(ctx, arg0, 803, 0));
  set(ctx, arg0, 804, get(ctx, arg0, 804, 0));
  set(ctx, arg0, 805, get(ctx, arg0, 805, 0));
  set(ctx, arg0, 806, get(ctx, arg0, 806, 0));
  set(ctx, arg0, 807, get(ctx, arg0, 807, 0));
  set(ctx, arg0, 808, get(ctx, arg0, 808, 0));
  set(ctx, arg0, 809, get(ctx, arg0, 809, 0));
  set(ctx, arg0, 810, get(ctx, arg0, 810, 0));
  set(ctx, arg0, 811, get(ctx, arg0, 811, 0));
  set(ctx, arg0, 812, get(ctx, arg0, 812, 0));
  set(ctx, arg0, 813, get(ctx, arg0, 813, 0));
  set(ctx, arg0, 814, get(ctx, arg0, 814, 0));
  set(ctx, arg0, 815, get(ctx, arg0, 815, 0));
  // TopState(zirgen/circuit/keccak2/top.zir:40)
  set(ctx, arg0, 816, get(ctx, arg0, 816, 0));
  set(ctx, arg0, 817, get(ctx, arg0, 817, 0));
  set(ctx, arg0, 818, get(ctx, arg0, 818, 0));
  set(ctx, arg0, 819, get(ctx, arg0, 819, 0));
  set(ctx, arg0, 820, get(ctx, arg0, 820, 0));
  set(ctx, arg0, 821, get(ctx, arg0, 821, 0));
  set(ctx, arg0, 822, get(ctx, arg0, 822, 0));
  set(ctx, arg0, 823, get(ctx, arg0, 823, 0));
  set(ctx, arg0, 824, get(ctx, arg0, 824, 0));
  set(ctx, arg0, 825, get(ctx, arg0, 825, 0));
  set(ctx, arg0, 826, get(ctx, arg0, 826, 0));
  set(ctx, arg0, 827, get(ctx, arg0, 827, 0));
  set(ctx, arg0, 828, get(ctx, arg0, 828, 0));
  set(ctx, arg0, 829, get(ctx, arg0, 829, 0));
  set(ctx, arg0, 830, get(ctx, arg0, 830, 0));
  set(ctx, arg0, 831, get(ctx, arg0, 831, 0));
  set(ctx, arg0, 832, get(ctx, arg0, 832, 0));
  set(ctx, arg0, 833, get(ctx, arg0, 833, 0));
  set(ctx, arg0, 834, get(ctx, arg0, 834, 0));
  set(ctx, arg0, 835, get(ctx, arg0, 835, 0));
  set(ctx, arg0, 836, get(ctx, arg0, 836, 0));
  set(ctx, arg0, 837, get(ctx, arg0, 837, 0));
  set(ctx, arg0, 838, get(ctx, arg0, 838, 0));
  set(ctx, arg0, 839, get(ctx, arg0, 839, 0));
  set(ctx, arg0, 840, get(ctx, arg0, 840, 0));
  set(ctx, arg0, 841, get(ctx, arg0, 841, 0));
  set(ctx, arg0, 842, get(ctx, arg0, 842, 0));
  set(ctx, arg0, 843, get(ctx, arg0, 843, 0));
  set(ctx, arg0, 844, get(ctx, arg0, 844, 0));
  set(ctx, arg0, 845, get(ctx, arg0, 845, 0));
  set(ctx, arg0, 846, get(ctx, arg0, 846, 0));
  set(ctx, arg0, 847, get(ctx, arg0, 847, 0));
  set(ctx, arg0, 848, get(ctx, arg0, 848, 0));
  set(ctx, arg0, 849, get(ctx, arg0, 849, 0));
  set(ctx, arg0, 850, get(ctx, arg0, 850, 0));
  set(ctx, arg0, 851, get(ctx, arg0, 851, 0));
  set(ctx, arg0, 852, get(ctx, arg0, 852, 0));
  set(ctx, arg0, 853, get(ctx, arg0, 853, 0));
  set(ctx, arg0, 854, get(ctx, arg0, 854, 0));
  set(ctx, arg0, 855, get(ctx, arg0, 855, 0));
  set(ctx, arg0, 856, get(ctx, arg0, 856, 0));
  set(ctx, arg0, 857, get(ctx, arg0, 857, 0));
  set(ctx, arg0, 858, get(ctx, arg0, 858, 0));
  set(ctx, arg0, 859, get(ctx, arg0, 859, 0));
  set(ctx, arg0, 860, get(ctx, arg0, 860, 0));
  set(ctx, arg0, 861, get(ctx, arg0, 861, 0));
  set(ctx, arg0, 862, get(ctx, arg0, 862, 0));
  set(ctx, arg0, 863, get(ctx, arg0, 863, 0));
  set(ctx, arg0, 864, get(ctx, arg0, 864, 0));
  set(ctx, arg0, 865, get(ctx, arg0, 865, 0));
  set(ctx, arg0, 866, get(ctx, arg0, 866, 0));
  set(ctx, arg0, 867, get(ctx, arg0, 867, 0));
  set(ctx, arg0, 868, get(ctx, arg0, 868, 0));
  set(ctx, arg0, 869, get(ctx, arg0, 869, 0));
  set(ctx, arg0, 870, get(ctx, arg0, 870, 0));
  set(ctx, arg0, 871, get(ctx, arg0, 871, 0));
  set(ctx, arg0, 872, get(ctx, arg0, 872, 0));
  set(ctx, arg0, 873, get(ctx, arg0, 873, 0));
  set(ctx, arg0, 874, get(ctx, arg0, 874, 0));
  set(ctx, arg0, 875, get(ctx, arg0, 875, 0));
  set(ctx, arg0, 876, get(ctx, arg0, 876, 0));
  set(ctx, arg0, 877, get(ctx, arg0, 877, 0));
  set(ctx, arg0, 878, get(ctx, arg0, 878, 0));
  set(ctx, arg0, 879, get(ctx, arg0, 879, 0));
  set(ctx, arg0, 880, get(ctx, arg0, 880, 0));
  set(ctx, arg0, 881, get(ctx, arg0, 881, 0));
  set(ctx, arg0, 882, get(ctx, arg0, 882, 0));
  set(ctx, arg0, 883, get(ctx, arg0, 883, 0));
  set(ctx, arg0, 884, get(ctx, arg0, 884, 0));
  set(ctx, arg0, 885, get(ctx, arg0, 885, 0));
  set(ctx, arg0, 886, get(ctx, arg0, 886, 0));
  set(ctx, arg0, 887, get(ctx, arg0, 887, 0));
  set(ctx, arg0, 888, get(ctx, arg0, 888, 0));
  set(ctx, arg0, 889, get(ctx, arg0, 889, 0));
  set(ctx, arg0, 890, get(ctx, arg0, 890, 0));
  set(ctx, arg0, 891, get(ctx, arg0, 891, 0));
  set(ctx, arg0, 892, get(ctx, arg0, 892, 0));
  set(ctx, arg0, 893, get(ctx, arg0, 893, 0));
  set(ctx, arg0, 894, get(ctx, arg0, 894, 0));
  set(ctx, arg0, 895, get(ctx, arg0, 895, 0));
  set(ctx, arg0, 896, get(ctx, arg0, 896, 0));
  set(ctx, arg0, 897, get(ctx, arg0, 897, 0));
  set(ctx, arg0, 898, get(ctx, arg0, 898, 0));
  set(ctx, arg0, 899, get(ctx, arg0, 899, 0));
  set(ctx, arg0, 900, get(ctx, arg0, 900, 0));
  set(ctx, arg0, 901, get(ctx, arg0, 901, 0));
  set(ctx, arg0, 902, get(ctx, arg0, 902, 0));
  set(ctx, arg0, 903, get(ctx, arg0, 903, 0));
  set(ctx, arg0, 904, get(ctx, arg0, 904, 0));
  set(ctx, arg0, 905, get(ctx, arg0, 905, 0));
  set(ctx, arg0, 906, get(ctx, arg0, 906, 0));
  set(ctx, arg0, 907, get(ctx, arg0, 907, 0));
  set(ctx, arg0, 908, get(ctx, arg0, 908, 0));
  set(ctx, arg0, 909, get(ctx, arg0, 909, 0));
  set(ctx, arg0, 910, get(ctx, arg0, 910, 0));
  set(ctx, arg0, 911, get(ctx, arg0, 911, 0));
  set(ctx, arg0, 912, get(ctx, arg0, 912, 0));
  set(ctx, arg0, 913, get(ctx, arg0, 913, 0));
  set(ctx, arg0, 914, get(ctx, arg0, 914, 0));
  set(ctx, arg0, 915, get(ctx, arg0, 915, 0));
  // TopState(zirgen/circuit/keccak2/top.zir:43)
  set(ctx, arg0, 916, x1);
  set(ctx, arg0, 917, x2);
  set(ctx, arg0, 918, x3);
  set(ctx, arg0, 919, x4);
  set(ctx, arg0, 920, x5);
  set(ctx, arg0, 921, x6);
  set(ctx, arg0, 922, x7);
  set(ctx, arg0, 923, x8);
  set(ctx, arg0, 924, x9);
  set(ctx, arg0, 925, x10);
  set(ctx, arg0, 926, x11);
  set(ctx, arg0, 927, x12);
  set(ctx, arg0, 928, x13);
  set(ctx, arg0, 929, x14);
  set(ctx, arg0, 930, x15);
  set(ctx, arg0, 931, x16);
  return;
=======
__device__ IotaStruct exec_Iota(ExecContext& ctx,
                                Val64Array5Array5Array arg0,
                                Val arg1,
                                BoundLayout<IotaLayout> layout2) {
  // Iota(zirgen/circuit/keccak2/keccak.zir:121)
  RoundToArrayStruct x3 = exec_RoundToArray(ctx, arg1, LAYOUT_LOOKUP(layout2, iotaArray));
  // BitXor(zirgen/circuit/keccak2/bits.zir:32)
  // Iota(zirgen/circuit/keccak2/keccak.zir:125)
  Val x4 = (x3._super[0]._super + arg0[0][0][0]);
  Val x5 = (x3._super[0]._super * Val(2));
  Val x6 = (x3._super[1]._super + arg0[0][0][1]);
  Val x7 = (x3._super[1]._super * Val(2));
  Val x8 = (x3._super[2]._super + arg0[0][0][2]);
  Val x9 = (x3._super[2]._super * Val(2));
  Val x10 = (x3._super[3]._super + arg0[0][0][3]);
  Val x11 = (x3._super[3]._super * Val(2));
  Val x12 = (x3._super[4]._super + arg0[0][0][4]);
  Val x13 = (x3._super[4]._super * Val(2));
  Val x14 = (x3._super[5]._super + arg0[0][0][5]);
  Val x15 = (x3._super[5]._super * Val(2));
  Val x16 = (x3._super[6]._super + arg0[0][0][6]);
  Val x17 = (x3._super[6]._super * Val(2));
  Val x18 = (x3._super[7]._super + arg0[0][0][7]);
  Val x19 = (x3._super[7]._super * Val(2));
  Val x20 = (x3._super[8]._super + arg0[0][0][8]);
  Val x21 = (x3._super[8]._super * Val(2));
  Val x22 = (x3._super[9]._super + arg0[0][0][9]);
  Val x23 = (x3._super[9]._super * Val(2));
  Val x24 = (x3._super[10]._super + arg0[0][0][10]);
  Val x25 = (x3._super[10]._super * Val(2));
  Val x26 = (x3._super[11]._super + arg0[0][0][11]);
  Val x27 = (x3._super[11]._super * Val(2));
  Val x28 = (x3._super[12]._super + arg0[0][0][12]);
  Val x29 = (x3._super[12]._super * Val(2));
  Val x30 = (x3._super[13]._super + arg0[0][0][13]);
  Val x31 = (x3._super[13]._super * Val(2));
  Val x32 = (x3._super[14]._super + arg0[0][0][14]);
  Val x33 = (x3._super[14]._super * Val(2));
  Val x34 = (x3._super[15]._super + arg0[0][0][15]);
  Val x35 = (x3._super[15]._super * Val(2));
  Val x36 = (x3._super[16]._super + arg0[0][0][16]);
  Val x37 = (x3._super[16]._super * Val(2));
  Val x38 = (x3._super[17]._super + arg0[0][0][17]);
  Val x39 = (x3._super[17]._super * Val(2));
  Val x40 = (x3._super[18]._super + arg0[0][0][18]);
  Val x41 = (x3._super[18]._super * Val(2));
  Val x42 = (x3._super[19]._super + arg0[0][0][19]);
  Val x43 = (x3._super[19]._super * Val(2));
  Val x44 = (x3._super[20]._super + arg0[0][0][20]);
  Val x45 = (x3._super[20]._super * Val(2));
  Val x46 = (x3._super[21]._super + arg0[0][0][21]);
  Val x47 = (x3._super[21]._super * Val(2));
  Val x48 = (x3._super[22]._super + arg0[0][0][22]);
  Val x49 = (x3._super[22]._super * Val(2));
  Val x50 = (x3._super[23]._super + arg0[0][0][23]);
  Val x51 = (x3._super[23]._super * Val(2));
  Val x52 = (x3._super[24]._super + arg0[0][0][24]);
  Val x53 = (x3._super[24]._super * Val(2));
  Val x54 = (x3._super[25]._super + arg0[0][0][25]);
  Val x55 = (x3._super[25]._super * Val(2));
  Val x56 = (x3._super[26]._super + arg0[0][0][26]);
  Val x57 = (x3._super[26]._super * Val(2));
  Val x58 = (x3._super[27]._super + arg0[0][0][27]);
  Val x59 = (x3._super[27]._super * Val(2));
  Val x60 = (x3._super[28]._super + arg0[0][0][28]);
  Val x61 = (x3._super[28]._super * Val(2));
  Val x62 = (x3._super[29]._super + arg0[0][0][29]);
  Val x63 = (x3._super[29]._super * Val(2));
  Val x64 = (x3._super[30]._super + arg0[0][0][30]);
  Val x65 = (x3._super[30]._super * Val(2));
  Val x66 = (x3._super[31]._super + arg0[0][0][31]);
  Val x67 = (x3._super[31]._super * Val(2));
  Val x68 = (x3._super[32]._super + arg0[0][0][32]);
  Val x69 = (x3._super[32]._super * Val(2));
  Val x70 = (x3._super[33]._super + arg0[0][0][33]);
  Val x71 = (x3._super[33]._super * Val(2));
  Val x72 = (x3._super[34]._super + arg0[0][0][34]);
  Val x73 = (x3._super[34]._super * Val(2));
  Val x74 = (x3._super[35]._super + arg0[0][0][35]);
  Val x75 = (x3._super[35]._super * Val(2));
  Val x76 = (x3._super[36]._super + arg0[0][0][36]);
  Val x77 = (x3._super[36]._super * Val(2));
  Val x78 = (x3._super[37]._super + arg0[0][0][37]);
  Val x79 = (x3._super[37]._super * Val(2));
  Val x80 = (x3._super[38]._super + arg0[0][0][38]);
  Val x81 = (x3._super[38]._super * Val(2));
  Val x82 = (x3._super[39]._super + arg0[0][0][39]);
  Val x83 = (x3._super[39]._super * Val(2));
  Val x84 = (x3._super[40]._super + arg0[0][0][40]);
  Val x85 = (x3._super[40]._super * Val(2));
  Val x86 = (x3._super[41]._super + arg0[0][0][41]);
  Val x87 = (x3._super[41]._super * Val(2));
  Val x88 = (x3._super[42]._super + arg0[0][0][42]);
  Val x89 = (x3._super[42]._super * Val(2));
  Val x90 = (x3._super[43]._super + arg0[0][0][43]);
  Val x91 = (x3._super[43]._super * Val(2));
  Val x92 = (x3._super[44]._super + arg0[0][0][44]);
  Val x93 = (x3._super[44]._super * Val(2));
  Val x94 = (x3._super[45]._super + arg0[0][0][45]);
  Val x95 = (x3._super[45]._super * Val(2));
  Val x96 = (x3._super[46]._super + arg0[0][0][46]);
  Val x97 = (x3._super[46]._super * Val(2));
  Val x98 = (x3._super[47]._super + arg0[0][0][47]);
  Val x99 = (x3._super[47]._super * Val(2));
  Val x100 = (x3._super[48]._super + arg0[0][0][48]);
  Val x101 = (x3._super[48]._super * Val(2));
  Val x102 = (x3._super[49]._super + arg0[0][0][49]);
  Val x103 = (x3._super[49]._super * Val(2));
  Val x104 = (x3._super[50]._super + arg0[0][0][50]);
  Val x105 = (x3._super[50]._super * Val(2));
  Val x106 = (x3._super[51]._super + arg0[0][0][51]);
  Val x107 = (x3._super[51]._super * Val(2));
  Val x108 = (x3._super[52]._super + arg0[0][0][52]);
  Val x109 = (x3._super[52]._super * Val(2));
  Val x110 = (x3._super[53]._super + arg0[0][0][53]);
  Val x111 = (x3._super[53]._super * Val(2));
  Val x112 = (x3._super[54]._super + arg0[0][0][54]);
  Val x113 = (x3._super[54]._super * Val(2));
  Val x114 = (x3._super[55]._super + arg0[0][0][55]);
  Val x115 = (x3._super[55]._super * Val(2));
  Val x116 = (x3._super[56]._super + arg0[0][0][56]);
  Val x117 = (x3._super[56]._super * Val(2));
  Val x118 = (x3._super[57]._super + arg0[0][0][57]);
  Val x119 = (x3._super[57]._super * Val(2));
  Val x120 = (x3._super[58]._super + arg0[0][0][58]);
  Val x121 = (x3._super[58]._super * Val(2));
  Val x122 = (x3._super[59]._super + arg0[0][0][59]);
  Val x123 = (x3._super[59]._super * Val(2));
  Val x124 = (x3._super[60]._super + arg0[0][0][60]);
  Val x125 = (x3._super[60]._super * Val(2));
  Val x126 = (x3._super[61]._super + arg0[0][0][61]);
  Val x127 = (x3._super[61]._super * Val(2));
  Val x128 = (x3._super[62]._super + arg0[0][0][62]);
  Val x129 = (x3._super[62]._super * Val(2));
  Val x130 = (x3._super[63]._super + arg0[0][0][63]);
  Val x131 = (x3._super[63]._super * Val(2));
  // Iota(zirgen/circuit/keccak2/keccak.zir:123)
  Iota_Super_Super_Super_SuperStruct x132 = Iota_Super_Super_Super_SuperStruct{
      ._super = Val64Array{(x4 - (x5 * arg0[0][0][0])),      (x6 - (x7 * arg0[0][0][1])),
                           (x8 - (x9 * arg0[0][0][2])),      (x10 - (x11 * arg0[0][0][3])),
                           (x12 - (x13 * arg0[0][0][4])),    (x14 - (x15 * arg0[0][0][5])),
                           (x16 - (x17 * arg0[0][0][6])),    (x18 - (x19 * arg0[0][0][7])),
                           (x20 - (x21 * arg0[0][0][8])),    (x22 - (x23 * arg0[0][0][9])),
                           (x24 - (x25 * arg0[0][0][10])),   (x26 - (x27 * arg0[0][0][11])),
                           (x28 - (x29 * arg0[0][0][12])),   (x30 - (x31 * arg0[0][0][13])),
                           (x32 - (x33 * arg0[0][0][14])),   (x34 - (x35 * arg0[0][0][15])),
                           (x36 - (x37 * arg0[0][0][16])),   (x38 - (x39 * arg0[0][0][17])),
                           (x40 - (x41 * arg0[0][0][18])),   (x42 - (x43 * arg0[0][0][19])),
                           (x44 - (x45 * arg0[0][0][20])),   (x46 - (x47 * arg0[0][0][21])),
                           (x48 - (x49 * arg0[0][0][22])),   (x50 - (x51 * arg0[0][0][23])),
                           (x52 - (x53 * arg0[0][0][24])),   (x54 - (x55 * arg0[0][0][25])),
                           (x56 - (x57 * arg0[0][0][26])),   (x58 - (x59 * arg0[0][0][27])),
                           (x60 - (x61 * arg0[0][0][28])),   (x62 - (x63 * arg0[0][0][29])),
                           (x64 - (x65 * arg0[0][0][30])),   (x66 - (x67 * arg0[0][0][31])),
                           (x68 - (x69 * arg0[0][0][32])),   (x70 - (x71 * arg0[0][0][33])),
                           (x72 - (x73 * arg0[0][0][34])),   (x74 - (x75 * arg0[0][0][35])),
                           (x76 - (x77 * arg0[0][0][36])),   (x78 - (x79 * arg0[0][0][37])),
                           (x80 - (x81 * arg0[0][0][38])),   (x82 - (x83 * arg0[0][0][39])),
                           (x84 - (x85 * arg0[0][0][40])),   (x86 - (x87 * arg0[0][0][41])),
                           (x88 - (x89 * arg0[0][0][42])),   (x90 - (x91 * arg0[0][0][43])),
                           (x92 - (x93 * arg0[0][0][44])),   (x94 - (x95 * arg0[0][0][45])),
                           (x96 - (x97 * arg0[0][0][46])),   (x98 - (x99 * arg0[0][0][47])),
                           (x100 - (x101 * arg0[0][0][48])), (x102 - (x103 * arg0[0][0][49])),
                           (x104 - (x105 * arg0[0][0][50])), (x106 - (x107 * arg0[0][0][51])),
                           (x108 - (x109 * arg0[0][0][52])), (x110 - (x111 * arg0[0][0][53])),
                           (x112 - (x113 * arg0[0][0][54])), (x114 - (x115 * arg0[0][0][55])),
                           (x116 - (x117 * arg0[0][0][56])), (x118 - (x119 * arg0[0][0][57])),
                           (x120 - (x121 * arg0[0][0][58])), (x122 - (x123 * arg0[0][0][59])),
                           (x124 - (x125 * arg0[0][0][60])), (x126 - (x127 * arg0[0][0][61])),
                           (x128 - (x129 * arg0[0][0][62])), (x130 - (x131 * arg0[0][0][63]))}};
  Iota_Super_Super_Super_SuperStruct x133 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[0][1]};
  Iota_Super_Super_Super_SuperStruct x134 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[0][2]};
  Iota_Super_Super_Super_SuperStruct x135 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[0][3]};
  Iota_Super_Super_Super_SuperStruct x136 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[0][4]};
  Iota_Super_Super_Super_SuperStruct x137 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[1][0]};
  Iota_Super_Super_Super_SuperStruct x138 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[1][1]};
  Iota_Super_Super_Super_SuperStruct x139 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[1][2]};
  Iota_Super_Super_Super_SuperStruct x140 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[1][3]};
  Iota_Super_Super_Super_SuperStruct x141 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[1][4]};
  Iota_Super_Super_Super_SuperStruct x142 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[2][0]};
  Iota_Super_Super_Super_SuperStruct x143 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[2][1]};
  Iota_Super_Super_Super_SuperStruct x144 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[2][2]};
  Iota_Super_Super_Super_SuperStruct x145 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[2][3]};
  Iota_Super_Super_Super_SuperStruct x146 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[2][4]};
  Iota_Super_Super_Super_SuperStruct x147 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[3][0]};
  Iota_Super_Super_Super_SuperStruct x148 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[3][1]};
  Iota_Super_Super_Super_SuperStruct x149 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[3][2]};
  Iota_Super_Super_Super_SuperStruct x150 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[3][3]};
  Iota_Super_Super_Super_SuperStruct x151 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[3][4]};
  Iota_Super_Super_Super_SuperStruct x152 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[4][0]};
  Iota_Super_Super_Super_SuperStruct x153 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[4][1]};
  Iota_Super_Super_Super_SuperStruct x154 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[4][2]};
  Iota_Super_Super_Super_SuperStruct x155 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[4][3]};
  Iota_Super_Super_Super_SuperStruct x156 =
      Iota_Super_Super_Super_SuperStruct{._super = arg0[4][4]};
  // Iota(zirgen/circuit/keccak2/keccak.zir:122)
  Iota_Super_SuperStruct5Array x157 = Iota_Super_SuperStruct5Array{
      Iota_Super_SuperStruct{
          ._super = Iota_Super_Super_Super_SuperStruct5Array{x132, x133, x134, x135, x136}},
      Iota_Super_SuperStruct{
          ._super = Iota_Super_Super_Super_SuperStruct5Array{x137, x138, x139, x140, x141}},
      Iota_Super_SuperStruct{
          ._super = Iota_Super_Super_Super_SuperStruct5Array{x142, x143, x144, x145, x146}},
      Iota_Super_SuperStruct{
          ._super = Iota_Super_Super_Super_SuperStruct5Array{x147, x148, x149, x150, x151}},
      Iota_Super_SuperStruct{
          ._super = Iota_Super_Super_Super_SuperStruct5Array{x152, x153, x154, x155, x156}}};
  return IotaStruct{._super = x157};
}
__device__ UnpackReg_800__16_Struct
exec_SFlatToBits(ExecContext& ctx, Val16Array arg0, BoundLayout<UnpackReg_800__16_Layout> layout1) {
  // SFlatToBits(zirgen/circuit/keccak2/top.zir:327)
  UnpackReg_800__16_Struct x2 = exec_UnpackReg_800__16_(
      ctx,
      Val50Array{Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),  Val(0),
                 arg0[6],  arg0[7],  arg0[4],  arg0[5],  arg0[2],  arg0[3],  arg0[0], arg0[1],
                 Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),  Val(0),
                 arg0[14], arg0[15], arg0[12], arg0[13], arg0[10], arg0[11], arg0[8], arg0[9],
                 Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),  Val(0),
                 Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),   Val(0),  Val(0),
                 Val(0),   Val(0)},
      layout1);
  return x2;
}
__device__ TopStateStruct exec_KeccakRound12_1_(ExecContext& ctx,
                                                TopStateStruct arg0,
                                                TopStateStruct arg1,
                                                TopStateStruct arg2,
                                                TopStateStruct arg3,
                                                BoundLayout<TopStateLayout> layout4) {
  // Log(<preamble>:22)
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:108)
  INVOKE_EXTERN(ctx, log, "KeccakRound12", std::initializer_list<Val>{});
  // BitXor(zirgen/circuit/keccak2/bits.zir:32)
  // ThetaP2(zirgen/circuit/keccak2/keccak.zir:25)
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:111)
  Val x5 = (arg0.bits[256]._super._super + arg0.bits[127]._super._super);
  Val x6 = (arg0.bits[256]._super._super * Val(2));
  Val x7 = (x5 - (x6 * arg0.bits[127]._super._super));
  Val x8 = (arg0.bits[257]._super._super + arg0.bits[64]._super._super);
  Val x9 = (arg0.bits[257]._super._super * Val(2));
  Val x10 = (x8 - (x9 * arg0.bits[64]._super._super));
  Val x11 = (arg0.bits[258]._super._super + arg0.bits[65]._super._super);
  Val x12 = (arg0.bits[258]._super._super * Val(2));
  Val x13 = (x11 - (x12 * arg0.bits[65]._super._super));
  Val x14 = (arg0.bits[259]._super._super + arg0.bits[66]._super._super);
  Val x15 = (arg0.bits[259]._super._super * Val(2));
  Val x16 = (x14 - (x15 * arg0.bits[66]._super._super));
  Val x17 = (arg0.bits[260]._super._super + arg0.bits[67]._super._super);
  Val x18 = (arg0.bits[260]._super._super * Val(2));
  Val x19 = (x17 - (x18 * arg0.bits[67]._super._super));
  Val x20 = (arg0.bits[261]._super._super + arg0.bits[68]._super._super);
  Val x21 = (arg0.bits[261]._super._super * Val(2));
  Val x22 = (x20 - (x21 * arg0.bits[68]._super._super));
  Val x23 = (arg0.bits[262]._super._super + arg0.bits[69]._super._super);
  Val x24 = (arg0.bits[262]._super._super * Val(2));
  Val x25 = (x23 - (x24 * arg0.bits[69]._super._super));
  Val x26 = (arg0.bits[263]._super._super + arg0.bits[70]._super._super);
  Val x27 = (arg0.bits[263]._super._super * Val(2));
  Val x28 = (x26 - (x27 * arg0.bits[70]._super._super));
  Val x29 = (arg0.bits[264]._super._super + arg0.bits[71]._super._super);
  Val x30 = (arg0.bits[264]._super._super * Val(2));
  Val x31 = (x29 - (x30 * arg0.bits[71]._super._super));
  Val x32 = (arg0.bits[265]._super._super + arg0.bits[72]._super._super);
  Val x33 = (arg0.bits[265]._super._super * Val(2));
  Val x34 = (x32 - (x33 * arg0.bits[72]._super._super));
  Val x35 = (arg0.bits[266]._super._super + arg0.bits[73]._super._super);
  Val x36 = (arg0.bits[266]._super._super * Val(2));
  Val x37 = (x35 - (x36 * arg0.bits[73]._super._super));
  Val x38 = (arg0.bits[267]._super._super + arg0.bits[74]._super._super);
  Val x39 = (arg0.bits[267]._super._super * Val(2));
  Val x40 = (x38 - (x39 * arg0.bits[74]._super._super));
  Val x41 = (arg0.bits[268]._super._super + arg0.bits[75]._super._super);
  Val x42 = (arg0.bits[268]._super._super * Val(2));
  Val x43 = (x41 - (x42 * arg0.bits[75]._super._super));
  Val x44 = (arg0.bits[269]._super._super + arg0.bits[76]._super._super);
  Val x45 = (arg0.bits[269]._super._super * Val(2));
  Val x46 = (x44 - (x45 * arg0.bits[76]._super._super));
  Val x47 = (arg0.bits[270]._super._super + arg0.bits[77]._super._super);
  Val x48 = (arg0.bits[270]._super._super * Val(2));
  Val x49 = (x47 - (x48 * arg0.bits[77]._super._super));
  Val x50 = (arg0.bits[271]._super._super + arg0.bits[78]._super._super);
  Val x51 = (arg0.bits[271]._super._super * Val(2));
  Val x52 = (x50 - (x51 * arg0.bits[78]._super._super));
  Val x53 = (arg0.bits[272]._super._super + arg0.bits[79]._super._super);
  Val x54 = (arg0.bits[272]._super._super * Val(2));
  Val x55 = (x53 - (x54 * arg0.bits[79]._super._super));
  Val x56 = (arg0.bits[273]._super._super + arg0.bits[80]._super._super);
  Val x57 = (arg0.bits[273]._super._super * Val(2));
  Val x58 = (x56 - (x57 * arg0.bits[80]._super._super));
  Val x59 = (arg0.bits[274]._super._super + arg0.bits[81]._super._super);
  Val x60 = (arg0.bits[274]._super._super * Val(2));
  Val x61 = (x59 - (x60 * arg0.bits[81]._super._super));
  Val x62 = (arg0.bits[275]._super._super + arg0.bits[82]._super._super);
  Val x63 = (arg0.bits[275]._super._super * Val(2));
  Val x64 = (x62 - (x63 * arg0.bits[82]._super._super));
  Val x65 = (arg0.bits[276]._super._super + arg0.bits[83]._super._super);
  Val x66 = (arg0.bits[276]._super._super * Val(2));
  Val x67 = (x65 - (x66 * arg0.bits[83]._super._super));
  Val x68 = (arg0.bits[277]._super._super + arg0.bits[84]._super._super);
  Val x69 = (arg0.bits[277]._super._super * Val(2));
  Val x70 = (x68 - (x69 * arg0.bits[84]._super._super));
  Val x71 = (arg0.bits[278]._super._super + arg0.bits[85]._super._super);
  Val x72 = (arg0.bits[278]._super._super * Val(2));
  Val x73 = (x71 - (x72 * arg0.bits[85]._super._super));
  Val x74 = (arg0.bits[279]._super._super + arg0.bits[86]._super._super);
  Val x75 = (arg0.bits[279]._super._super * Val(2));
  Val x76 = (x74 - (x75 * arg0.bits[86]._super._super));
  Val x77 = (arg0.bits[280]._super._super + arg0.bits[87]._super._super);
  Val x78 = (arg0.bits[280]._super._super * Val(2));
  Val x79 = (x77 - (x78 * arg0.bits[87]._super._super));
  Val x80 = (arg0.bits[281]._super._super + arg0.bits[88]._super._super);
  Val x81 = (arg0.bits[281]._super._super * Val(2));
  Val x82 = (x80 - (x81 * arg0.bits[88]._super._super));
  Val x83 = (arg0.bits[282]._super._super + arg0.bits[89]._super._super);
  Val x84 = (arg0.bits[282]._super._super * Val(2));
  Val x85 = (x83 - (x84 * arg0.bits[89]._super._super));
  Val x86 = (arg0.bits[283]._super._super + arg0.bits[90]._super._super);
  Val x87 = (arg0.bits[283]._super._super * Val(2));
  Val x88 = (x86 - (x87 * arg0.bits[90]._super._super));
  Val x89 = (arg0.bits[284]._super._super + arg0.bits[91]._super._super);
  Val x90 = (arg0.bits[284]._super._super * Val(2));
  Val x91 = (x89 - (x90 * arg0.bits[91]._super._super));
  Val x92 = (arg0.bits[285]._super._super + arg0.bits[92]._super._super);
  Val x93 = (arg0.bits[285]._super._super * Val(2));
  Val x94 = (x92 - (x93 * arg0.bits[92]._super._super));
  Val x95 = (arg0.bits[286]._super._super + arg0.bits[93]._super._super);
  Val x96 = (arg0.bits[286]._super._super * Val(2));
  Val x97 = (x95 - (x96 * arg0.bits[93]._super._super));
  Val x98 = (arg0.bits[287]._super._super + arg0.bits[94]._super._super);
  Val x99 = (arg0.bits[287]._super._super * Val(2));
  Val x100 = (x98 - (x99 * arg0.bits[94]._super._super));
  Val x101 = (arg0.bits[288]._super._super + arg0.bits[95]._super._super);
  Val x102 = (arg0.bits[288]._super._super * Val(2));
  Val x103 = (x101 - (x102 * arg0.bits[95]._super._super));
  Val x104 = (arg0.bits[289]._super._super + arg0.bits[96]._super._super);
  Val x105 = (arg0.bits[289]._super._super * Val(2));
  Val x106 = (x104 - (x105 * arg0.bits[96]._super._super));
  Val x107 = (arg0.bits[290]._super._super + arg0.bits[97]._super._super);
  Val x108 = (arg0.bits[290]._super._super * Val(2));
  Val x109 = (x107 - (x108 * arg0.bits[97]._super._super));
  Val x110 = (arg0.bits[291]._super._super + arg0.bits[98]._super._super);
  Val x111 = (arg0.bits[291]._super._super * Val(2));
  Val x112 = (x110 - (x111 * arg0.bits[98]._super._super));
  Val x113 = (arg0.bits[292]._super._super + arg0.bits[99]._super._super);
  Val x114 = (arg0.bits[292]._super._super * Val(2));
  Val x115 = (x113 - (x114 * arg0.bits[99]._super._super));
  Val x116 = (arg0.bits[293]._super._super + arg0.bits[100]._super._super);
  Val x117 = (arg0.bits[293]._super._super * Val(2));
  Val x118 = (x116 - (x117 * arg0.bits[100]._super._super));
  Val x119 = (arg0.bits[294]._super._super + arg0.bits[101]._super._super);
  Val x120 = (arg0.bits[294]._super._super * Val(2));
  Val x121 = (x119 - (x120 * arg0.bits[101]._super._super));
  Val x122 = (arg0.bits[295]._super._super + arg0.bits[102]._super._super);
  Val x123 = (arg0.bits[295]._super._super * Val(2));
  Val x124 = (x122 - (x123 * arg0.bits[102]._super._super));
  Val x125 = (arg0.bits[296]._super._super + arg0.bits[103]._super._super);
  Val x126 = (arg0.bits[296]._super._super * Val(2));
  Val x127 = (x125 - (x126 * arg0.bits[103]._super._super));
  Val x128 = (arg0.bits[297]._super._super + arg0.bits[104]._super._super);
  Val x129 = (arg0.bits[297]._super._super * Val(2));
  Val x130 = (x128 - (x129 * arg0.bits[104]._super._super));
  Val x131 = (arg0.bits[298]._super._super + arg0.bits[105]._super._super);
  Val x132 = (arg0.bits[298]._super._super * Val(2));
  Val x133 = (x131 - (x132 * arg0.bits[105]._super._super));
  Val x134 = (arg0.bits[299]._super._super + arg0.bits[106]._super._super);
  Val x135 = (arg0.bits[299]._super._super * Val(2));
  Val x136 = (x134 - (x135 * arg0.bits[106]._super._super));
  Val x137 = (arg0.bits[300]._super._super + arg0.bits[107]._super._super);
  Val x138 = (arg0.bits[300]._super._super * Val(2));
  Val x139 = (x137 - (x138 * arg0.bits[107]._super._super));
  Val x140 = (arg0.bits[301]._super._super + arg0.bits[108]._super._super);
  Val x141 = (arg0.bits[301]._super._super * Val(2));
  Val x142 = (x140 - (x141 * arg0.bits[108]._super._super));
  Val x143 = (arg0.bits[302]._super._super + arg0.bits[109]._super._super);
  Val x144 = (arg0.bits[302]._super._super * Val(2));
  Val x145 = (x143 - (x144 * arg0.bits[109]._super._super));
  Val x146 = (arg0.bits[303]._super._super + arg0.bits[110]._super._super);
  Val x147 = (arg0.bits[303]._super._super * Val(2));
  Val x148 = (x146 - (x147 * arg0.bits[110]._super._super));
  Val x149 = (arg0.bits[304]._super._super + arg0.bits[111]._super._super);
  Val x150 = (arg0.bits[304]._super._super * Val(2));
  Val x151 = (x149 - (x150 * arg0.bits[111]._super._super));
  Val x152 = (arg0.bits[305]._super._super + arg0.bits[112]._super._super);
  Val x153 = (arg0.bits[305]._super._super * Val(2));
  Val x154 = (x152 - (x153 * arg0.bits[112]._super._super));
  Val x155 = (arg0.bits[306]._super._super + arg0.bits[113]._super._super);
  Val x156 = (arg0.bits[306]._super._super * Val(2));
  Val x157 = (x155 - (x156 * arg0.bits[113]._super._super));
  Val x158 = (arg0.bits[307]._super._super + arg0.bits[114]._super._super);
  Val x159 = (arg0.bits[307]._super._super * Val(2));
  Val x160 = (x158 - (x159 * arg0.bits[114]._super._super));
  Val x161 = (arg0.bits[308]._super._super + arg0.bits[115]._super._super);
  Val x162 = (arg0.bits[308]._super._super * Val(2));
  Val x163 = (x161 - (x162 * arg0.bits[115]._super._super));
  Val x164 = (arg0.bits[309]._super._super + arg0.bits[116]._super._super);
  Val x165 = (arg0.bits[309]._super._super * Val(2));
  Val x166 = (x164 - (x165 * arg0.bits[116]._super._super));
  Val x167 = (arg0.bits[310]._super._super + arg0.bits[117]._super._super);
  Val x168 = (arg0.bits[310]._super._super * Val(2));
  Val x169 = (x167 - (x168 * arg0.bits[117]._super._super));
  Val x170 = (arg0.bits[311]._super._super + arg0.bits[118]._super._super);
  Val x171 = (arg0.bits[311]._super._super * Val(2));
  Val x172 = (x170 - (x171 * arg0.bits[118]._super._super));
  Val x173 = (arg0.bits[312]._super._super + arg0.bits[119]._super._super);
  Val x174 = (arg0.bits[312]._super._super * Val(2));
  Val x175 = (x173 - (x174 * arg0.bits[119]._super._super));
  Val x176 = (arg0.bits[313]._super._super + arg0.bits[120]._super._super);
  Val x177 = (arg0.bits[313]._super._super * Val(2));
  Val x178 = (x176 - (x177 * arg0.bits[120]._super._super));
  Val x179 = (arg0.bits[314]._super._super + arg0.bits[121]._super._super);
  Val x180 = (arg0.bits[314]._super._super * Val(2));
  Val x181 = (x179 - (x180 * arg0.bits[121]._super._super));
  Val x182 = (arg0.bits[315]._super._super + arg0.bits[122]._super._super);
  Val x183 = (arg0.bits[315]._super._super * Val(2));
  Val x184 = (x182 - (x183 * arg0.bits[122]._super._super));
  Val x185 = (arg0.bits[316]._super._super + arg0.bits[123]._super._super);
  Val x186 = (arg0.bits[316]._super._super * Val(2));
  Val x187 = (x185 - (x186 * arg0.bits[123]._super._super));
  Val x188 = (arg0.bits[317]._super._super + arg0.bits[124]._super._super);
  Val x189 = (arg0.bits[317]._super._super * Val(2));
  Val x190 = (x188 - (x189 * arg0.bits[124]._super._super));
  Val x191 = (arg0.bits[318]._super._super + arg0.bits[125]._super._super);
  Val x192 = (arg0.bits[318]._super._super * Val(2));
  Val x193 = (x191 - (x192 * arg0.bits[125]._super._super));
  Val x194 = (arg0.bits[319]._super._super + arg0.bits[126]._super._super);
  Val x195 = (arg0.bits[319]._super._super * Val(2));
  Val x196 = (x194 - (x195 * arg0.bits[126]._super._super));
  Val x197 = (arg0.bits[0]._super._super + arg0.bits[191]._super._super);
  Val x198 = (arg0.bits[0]._super._super * Val(2));
  Val x199 = (x197 - (x198 * arg0.bits[191]._super._super));
  Val x200 = (arg0.bits[1]._super._super + arg0.bits[128]._super._super);
  Val x201 = (arg0.bits[1]._super._super * Val(2));
  Val x202 = (x200 - (x201 * arg0.bits[128]._super._super));
  Val x203 = (arg0.bits[2]._super._super + arg0.bits[129]._super._super);
  Val x204 = (arg0.bits[2]._super._super * Val(2));
  Val x205 = (x203 - (x204 * arg0.bits[129]._super._super));
  Val x206 = (arg0.bits[3]._super._super + arg0.bits[130]._super._super);
  Val x207 = (arg0.bits[3]._super._super * Val(2));
  Val x208 = (x206 - (x207 * arg0.bits[130]._super._super));
  Val x209 = (arg0.bits[4]._super._super + arg0.bits[131]._super._super);
  Val x210 = (arg0.bits[4]._super._super * Val(2));
  Val x211 = (x209 - (x210 * arg0.bits[131]._super._super));
  Val x212 = (arg0.bits[5]._super._super + arg0.bits[132]._super._super);
  Val x213 = (arg0.bits[5]._super._super * Val(2));
  Val x214 = (x212 - (x213 * arg0.bits[132]._super._super));
  Val x215 = (arg0.bits[6]._super._super + arg0.bits[133]._super._super);
  Val x216 = (arg0.bits[6]._super._super * Val(2));
  Val x217 = (x215 - (x216 * arg0.bits[133]._super._super));
  Val x218 = (arg0.bits[7]._super._super + arg0.bits[134]._super._super);
  Val x219 = (arg0.bits[7]._super._super * Val(2));
  Val x220 = (x218 - (x219 * arg0.bits[134]._super._super));
  Val x221 = (arg0.bits[8]._super._super + arg0.bits[135]._super._super);
  Val x222 = (arg0.bits[8]._super._super * Val(2));
  Val x223 = (x221 - (x222 * arg0.bits[135]._super._super));
  Val x224 = (arg0.bits[9]._super._super + arg0.bits[136]._super._super);
  Val x225 = (arg0.bits[9]._super._super * Val(2));
  Val x226 = (x224 - (x225 * arg0.bits[136]._super._super));
  Val x227 = (arg0.bits[10]._super._super + arg0.bits[137]._super._super);
  Val x228 = (arg0.bits[10]._super._super * Val(2));
  Val x229 = (x227 - (x228 * arg0.bits[137]._super._super));
  Val x230 = (arg0.bits[11]._super._super + arg0.bits[138]._super._super);
  Val x231 = (arg0.bits[11]._super._super * Val(2));
  Val x232 = (x230 - (x231 * arg0.bits[138]._super._super));
  Val x233 = (arg0.bits[12]._super._super + arg0.bits[139]._super._super);
  Val x234 = (arg0.bits[12]._super._super * Val(2));
  Val x235 = (x233 - (x234 * arg0.bits[139]._super._super));
  Val x236 = (arg0.bits[13]._super._super + arg0.bits[140]._super._super);
  Val x237 = (arg0.bits[13]._super._super * Val(2));
  Val x238 = (x236 - (x237 * arg0.bits[140]._super._super));
  Val x239 = (arg0.bits[14]._super._super + arg0.bits[141]._super._super);
  Val x240 = (arg0.bits[14]._super._super * Val(2));
  Val x241 = (x239 - (x240 * arg0.bits[141]._super._super));
  Val x242 = (arg0.bits[15]._super._super + arg0.bits[142]._super._super);
  Val x243 = (arg0.bits[15]._super._super * Val(2));
  Val x244 = (x242 - (x243 * arg0.bits[142]._super._super));
  Val x245 = (arg0.bits[16]._super._super + arg0.bits[143]._super._super);
  Val x246 = (arg0.bits[16]._super._super * Val(2));
  Val x247 = (x245 - (x246 * arg0.bits[143]._super._super));
  Val x248 = (arg0.bits[17]._super._super + arg0.bits[144]._super._super);
  Val x249 = (arg0.bits[17]._super._super * Val(2));
  Val x250 = (x248 - (x249 * arg0.bits[144]._super._super));
  Val x251 = (arg0.bits[18]._super._super + arg0.bits[145]._super._super);
  Val x252 = (arg0.bits[18]._super._super * Val(2));
  Val x253 = (x251 - (x252 * arg0.bits[145]._super._super));
  Val x254 = (arg0.bits[19]._super._super + arg0.bits[146]._super._super);
  Val x255 = (arg0.bits[19]._super._super * Val(2));
  Val x256 = (x254 - (x255 * arg0.bits[146]._super._super));
  Val x257 = (arg0.bits[22]._super._super + arg0.bits[149]._super._super);
  Val x258 = (arg0.bits[22]._super._super * Val(2));
  Val x259 = (x257 - (x258 * arg0.bits[149]._super._super));
  Val x260 = (arg0.bits[23]._super._super + arg0.bits[150]._super._super);
  Val x261 = (arg0.bits[23]._super._super * Val(2));
  Val x262 = (x260 - (x261 * arg0.bits[150]._super._super));
  Val x263 = (arg0.bits[24]._super._super + arg0.bits[151]._super._super);
  Val x264 = (arg0.bits[24]._super._super * Val(2));
  Val x265 = (x263 - (x264 * arg0.bits[151]._super._super));
  Val x266 = (arg0.bits[25]._super._super + arg0.bits[152]._super._super);
  Val x267 = (arg0.bits[25]._super._super * Val(2));
  Val x268 = (x266 - (x267 * arg0.bits[152]._super._super));
  Val x269 = (arg0.bits[26]._super._super + arg0.bits[153]._super._super);
  Val x270 = (arg0.bits[26]._super._super * Val(2));
  Val x271 = (x269 - (x270 * arg0.bits[153]._super._super));
  Val x272 = (arg0.bits[27]._super._super + arg0.bits[154]._super._super);
  Val x273 = (arg0.bits[27]._super._super * Val(2));
  Val x274 = (x272 - (x273 * arg0.bits[154]._super._super));
  Val x275 = (arg0.bits[28]._super._super + arg0.bits[155]._super._super);
  Val x276 = (arg0.bits[28]._super._super * Val(2));
  Val x277 = (x275 - (x276 * arg0.bits[155]._super._super));
  Val x278 = (arg0.bits[29]._super._super + arg0.bits[156]._super._super);
  Val x279 = (arg0.bits[29]._super._super * Val(2));
  Val x280 = (x278 - (x279 * arg0.bits[156]._super._super));
  Val x281 = (arg0.bits[30]._super._super + arg0.bits[157]._super._super);
  Val x282 = (arg0.bits[30]._super._super * Val(2));
  Val x283 = (x281 - (x282 * arg0.bits[157]._super._super));
  Val x284 = (arg0.bits[31]._super._super + arg0.bits[158]._super._super);
  Val x285 = (arg0.bits[31]._super._super * Val(2));
  Val x286 = (x284 - (x285 * arg0.bits[158]._super._super));
  Val x287 = (arg0.bits[32]._super._super + arg0.bits[159]._super._super);
  Val x288 = (arg0.bits[32]._super._super * Val(2));
  Val x289 = (x287 - (x288 * arg0.bits[159]._super._super));
  Val x290 = (arg0.bits[33]._super._super + arg0.bits[160]._super._super);
  Val x291 = (arg0.bits[33]._super._super * Val(2));
  Val x292 = (x290 - (x291 * arg0.bits[160]._super._super));
  Val x293 = (arg0.bits[34]._super._super + arg0.bits[161]._super._super);
  Val x294 = (arg0.bits[34]._super._super * Val(2));
  Val x295 = (x293 - (x294 * arg0.bits[161]._super._super));
  Val x296 = (arg0.bits[35]._super._super + arg0.bits[162]._super._super);
  Val x297 = (arg0.bits[35]._super._super * Val(2));
  Val x298 = (x296 - (x297 * arg0.bits[162]._super._super));
  Val x299 = (arg0.bits[36]._super._super + arg0.bits[163]._super._super);
  Val x300 = (arg0.bits[36]._super._super * Val(2));
  Val x301 = (x299 - (x300 * arg0.bits[163]._super._super));
  Val x302 = (arg0.bits[37]._super._super + arg0.bits[164]._super._super);
  Val x303 = (arg0.bits[37]._super._super * Val(2));
  Val x304 = (x302 - (x303 * arg0.bits[164]._super._super));
  Val x305 = (arg0.bits[38]._super._super + arg0.bits[165]._super._super);
  Val x306 = (arg0.bits[38]._super._super * Val(2));
  Val x307 = (x305 - (x306 * arg0.bits[165]._super._super));
  Val x308 = (arg0.bits[39]._super._super + arg0.bits[166]._super._super);
  Val x309 = (arg0.bits[39]._super._super * Val(2));
  Val x310 = (x308 - (x309 * arg0.bits[166]._super._super));
  Val x311 = (arg0.bits[40]._super._super + arg0.bits[167]._super._super);
  Val x312 = (arg0.bits[40]._super._super * Val(2));
  Val x313 = (x311 - (x312 * arg0.bits[167]._super._super));
  Val x314 = (arg0.bits[41]._super._super + arg0.bits[168]._super._super);
  Val x315 = (arg0.bits[41]._super._super * Val(2));
  Val x316 = (x314 - (x315 * arg0.bits[168]._super._super));
  Val x317 = (arg0.bits[42]._super._super + arg0.bits[169]._super._super);
  Val x318 = (arg0.bits[42]._super._super * Val(2));
  Val x319 = (x317 - (x318 * arg0.bits[169]._super._super));
  Val x320 = (arg0.bits[43]._super._super + arg0.bits[170]._super._super);
  Val x321 = (arg0.bits[43]._super._super * Val(2));
  Val x322 = (x320 - (x321 * arg0.bits[170]._super._super));
  Val x323 = (arg0.bits[44]._super._super + arg0.bits[171]._super._super);
  Val x324 = (arg0.bits[44]._super._super * Val(2));
  Val x325 = (x323 - (x324 * arg0.bits[171]._super._super));
  Val x326 = (arg0.bits[45]._super._super + arg0.bits[172]._super._super);
  Val x327 = (arg0.bits[45]._super._super * Val(2));
  Val x328 = (x326 - (x327 * arg0.bits[172]._super._super));
  Val x329 = (arg0.bits[46]._super._super + arg0.bits[173]._super._super);
  Val x330 = (arg0.bits[46]._super._super * Val(2));
  Val x331 = (x329 - (x330 * arg0.bits[173]._super._super));
  Val x332 = (arg0.bits[47]._super._super + arg0.bits[174]._super._super);
  Val x333 = (arg0.bits[47]._super._super * Val(2));
  Val x334 = (x332 - (x333 * arg0.bits[174]._super._super));
  Val x335 = (arg0.bits[48]._super._super + arg0.bits[175]._super._super);
  Val x336 = (arg0.bits[48]._super._super * Val(2));
  Val x337 = (x335 - (x336 * arg0.bits[175]._super._super));
  Val x338 = (arg0.bits[49]._super._super + arg0.bits[176]._super._super);
  Val x339 = (arg0.bits[49]._super._super * Val(2));
  Val x340 = (x338 - (x339 * arg0.bits[176]._super._super));
  Val x341 = (arg0.bits[50]._super._super + arg0.bits[177]._super._super);
  Val x342 = (arg0.bits[50]._super._super * Val(2));
  Val x343 = (x341 - (x342 * arg0.bits[177]._super._super));
  Val x344 = (arg0.bits[51]._super._super + arg0.bits[178]._super._super);
  Val x345 = (arg0.bits[51]._super._super * Val(2));
  Val x346 = (x344 - (x345 * arg0.bits[178]._super._super));
  Val x347 = (arg0.bits[52]._super._super + arg0.bits[179]._super._super);
  Val x348 = (arg0.bits[52]._super._super * Val(2));
  Val x349 = (x347 - (x348 * arg0.bits[179]._super._super));
  Val x350 = (arg0.bits[53]._super._super + arg0.bits[180]._super._super);
  Val x351 = (arg0.bits[53]._super._super * Val(2));
  Val x352 = (x350 - (x351 * arg0.bits[180]._super._super));
  Val x353 = (arg0.bits[54]._super._super + arg0.bits[181]._super._super);
  Val x354 = (arg0.bits[54]._super._super * Val(2));
  Val x355 = (x353 - (x354 * arg0.bits[181]._super._super));
  Val x356 = (arg0.bits[55]._super._super + arg0.bits[182]._super._super);
  Val x357 = (arg0.bits[55]._super._super * Val(2));
  Val x358 = (x356 - (x357 * arg0.bits[182]._super._super));
  Val x359 = (arg0.bits[56]._super._super + arg0.bits[183]._super._super);
  Val x360 = (arg0.bits[56]._super._super * Val(2));
  Val x361 = (x359 - (x360 * arg0.bits[183]._super._super));
  Val x362 = (arg0.bits[57]._super._super + arg0.bits[184]._super._super);
  Val x363 = (arg0.bits[57]._super._super * Val(2));
  Val x364 = (x362 - (x363 * arg0.bits[184]._super._super));
  Val x365 = (arg0.bits[58]._super._super + arg0.bits[185]._super._super);
  Val x366 = (arg0.bits[58]._super._super * Val(2));
  Val x367 = (x365 - (x366 * arg0.bits[185]._super._super));
  Val x368 = (arg0.bits[59]._super._super + arg0.bits[186]._super._super);
  Val x369 = (arg0.bits[59]._super._super * Val(2));
  Val x370 = (x368 - (x369 * arg0.bits[186]._super._super));
  Val x371 = (arg0.bits[60]._super._super + arg0.bits[187]._super._super);
  Val x372 = (arg0.bits[60]._super._super * Val(2));
  Val x373 = (x371 - (x372 * arg0.bits[187]._super._super));
  Val x374 = (arg0.bits[61]._super._super + arg0.bits[188]._super._super);
  Val x375 = (arg0.bits[61]._super._super * Val(2));
  Val x376 = (x374 - (x375 * arg0.bits[188]._super._super));
  Val x377 = (arg0.bits[62]._super._super + arg0.bits[189]._super._super);
  Val x378 = (arg0.bits[62]._super._super * Val(2));
  Val x379 = (x377 - (x378 * arg0.bits[189]._super._super));
  Val x380 = (arg0.bits[63]._super._super + arg0.bits[190]._super._super);
  Val x381 = (arg0.bits[63]._super._super * Val(2));
  Val x382 = (x380 - (x381 * arg0.bits[190]._super._super));
  Val x383 = (arg0.bits[64]._super._super + arg0.bits[255]._super._super);
  Val x384 = (arg0.bits[64]._super._super * Val(2));
  Val x385 = (x383 - (x384 * arg0.bits[255]._super._super));
  Val x386 = (arg0.bits[65]._super._super + arg0.bits[192]._super._super);
  Val x387 = (arg0.bits[65]._super._super * Val(2));
  Val x388 = (x386 - (x387 * arg0.bits[192]._super._super));
  Val x389 = (arg0.bits[66]._super._super + arg0.bits[193]._super._super);
  Val x390 = (arg0.bits[66]._super._super * Val(2));
  Val x391 = (x389 - (x390 * arg0.bits[193]._super._super));
  Val x392 = (arg0.bits[67]._super._super + arg0.bits[194]._super._super);
  Val x393 = (arg0.bits[67]._super._super * Val(2));
  Val x394 = (x392 - (x393 * arg0.bits[194]._super._super));
  Val x395 = (arg0.bits[68]._super._super + arg0.bits[195]._super._super);
  Val x396 = (arg0.bits[68]._super._super * Val(2));
  Val x397 = (x395 - (x396 * arg0.bits[195]._super._super));
  Val x398 = (arg0.bits[69]._super._super + arg0.bits[196]._super._super);
  Val x399 = (arg0.bits[69]._super._super * Val(2));
  Val x400 = (x398 - (x399 * arg0.bits[196]._super._super));
  Val x401 = (arg0.bits[70]._super._super + arg0.bits[197]._super._super);
  Val x402 = (arg0.bits[70]._super._super * Val(2));
  Val x403 = (x401 - (x402 * arg0.bits[197]._super._super));
  Val x404 = (arg0.bits[71]._super._super + arg0.bits[198]._super._super);
  Val x405 = (arg0.bits[71]._super._super * Val(2));
  Val x406 = (x404 - (x405 * arg0.bits[198]._super._super));
  Val x407 = (arg0.bits[72]._super._super + arg0.bits[199]._super._super);
  Val x408 = (arg0.bits[72]._super._super * Val(2));
  Val x409 = (x407 - (x408 * arg0.bits[199]._super._super));
  Val x410 = (arg0.bits[73]._super._super + arg0.bits[200]._super._super);
  Val x411 = (arg0.bits[73]._super._super * Val(2));
  Val x412 = (x410 - (x411 * arg0.bits[200]._super._super));
  Val x413 = (arg0.bits[74]._super._super + arg0.bits[201]._super._super);
  Val x414 = (arg0.bits[74]._super._super * Val(2));
  Val x415 = (x413 - (x414 * arg0.bits[201]._super._super));
  Val x416 = (arg0.bits[75]._super._super + arg0.bits[202]._super._super);
  Val x417 = (arg0.bits[75]._super._super * Val(2));
  Val x418 = (x416 - (x417 * arg0.bits[202]._super._super));
  Val x419 = (arg0.bits[76]._super._super + arg0.bits[203]._super._super);
  Val x420 = (arg0.bits[76]._super._super * Val(2));
  Val x421 = (x419 - (x420 * arg0.bits[203]._super._super));
  Val x422 = (arg0.bits[77]._super._super + arg0.bits[204]._super._super);
  Val x423 = (arg0.bits[77]._super._super * Val(2));
  Val x424 = (x422 - (x423 * arg0.bits[204]._super._super));
  Val x425 = (arg0.bits[78]._super._super + arg0.bits[205]._super._super);
  Val x426 = (arg0.bits[78]._super._super * Val(2));
  Val x427 = (x425 - (x426 * arg0.bits[205]._super._super));
  Val x428 = (arg0.bits[79]._super._super + arg0.bits[206]._super._super);
  Val x429 = (arg0.bits[79]._super._super * Val(2));
  Val x430 = (x428 - (x429 * arg0.bits[206]._super._super));
  Val x431 = (arg0.bits[80]._super._super + arg0.bits[207]._super._super);
  Val x432 = (arg0.bits[80]._super._super * Val(2));
  Val x433 = (x431 - (x432 * arg0.bits[207]._super._super));
  Val x434 = (arg0.bits[81]._super._super + arg0.bits[208]._super._super);
  Val x435 = (arg0.bits[81]._super._super * Val(2));
  Val x436 = (x434 - (x435 * arg0.bits[208]._super._super));
  Val x437 = (arg0.bits[82]._super._super + arg0.bits[209]._super._super);
  Val x438 = (arg0.bits[82]._super._super * Val(2));
  Val x439 = (x437 - (x438 * arg0.bits[209]._super._super));
  Val x440 = (arg0.bits[83]._super._super + arg0.bits[210]._super._super);
  Val x441 = (arg0.bits[83]._super._super * Val(2));
  Val x442 = (x440 - (x441 * arg0.bits[210]._super._super));
  Val x443 = (arg0.bits[84]._super._super + arg0.bits[211]._super._super);
  Val x444 = (arg0.bits[84]._super._super * Val(2));
  Val x445 = (x443 - (x444 * arg0.bits[211]._super._super));
  Val x446 = (arg0.bits[85]._super._super + arg0.bits[212]._super._super);
  Val x447 = (arg0.bits[85]._super._super * Val(2));
  Val x448 = (x446 - (x447 * arg0.bits[212]._super._super));
  Val x449 = (arg0.bits[86]._super._super + arg0.bits[213]._super._super);
  Val x450 = (arg0.bits[86]._super._super * Val(2));
  Val x451 = (x449 - (x450 * arg0.bits[213]._super._super));
  Val x452 = (arg0.bits[87]._super._super + arg0.bits[214]._super._super);
  Val x453 = (arg0.bits[87]._super._super * Val(2));
  Val x454 = (x452 - (x453 * arg0.bits[214]._super._super));
  Val x455 = (arg0.bits[88]._super._super + arg0.bits[215]._super._super);
  Val x456 = (arg0.bits[88]._super._super * Val(2));
  Val x457 = (x455 - (x456 * arg0.bits[215]._super._super));
  Val x458 = (arg0.bits[89]._super._super + arg0.bits[216]._super._super);
  Val x459 = (arg0.bits[89]._super._super * Val(2));
  Val x460 = (x458 - (x459 * arg0.bits[216]._super._super));
  Val x461 = (arg0.bits[90]._super._super + arg0.bits[217]._super._super);
  Val x462 = (arg0.bits[90]._super._super * Val(2));
  Val x463 = (x461 - (x462 * arg0.bits[217]._super._super));
  Val x464 = (arg0.bits[91]._super._super + arg0.bits[218]._super._super);
  Val x465 = (arg0.bits[91]._super._super * Val(2));
  Val x466 = (x464 - (x465 * arg0.bits[218]._super._super));
  Val x467 = (arg0.bits[92]._super._super + arg0.bits[219]._super._super);
  Val x468 = (arg0.bits[92]._super._super * Val(2));
  Val x469 = (x467 - (x468 * arg0.bits[219]._super._super));
  Val x470 = (arg0.bits[93]._super._super + arg0.bits[220]._super._super);
  Val x471 = (arg0.bits[93]._super._super * Val(2));
  Val x472 = (x470 - (x471 * arg0.bits[220]._super._super));
  Val x473 = (arg0.bits[94]._super._super + arg0.bits[221]._super._super);
  Val x474 = (arg0.bits[94]._super._super * Val(2));
  Val x475 = (x473 - (x474 * arg0.bits[221]._super._super));
  Val x476 = (arg0.bits[95]._super._super + arg0.bits[222]._super._super);
  Val x477 = (arg0.bits[95]._super._super * Val(2));
  Val x478 = (x476 - (x477 * arg0.bits[222]._super._super));
  Val x479 = (arg0.bits[96]._super._super + arg0.bits[223]._super._super);
  Val x480 = (arg0.bits[96]._super._super * Val(2));
  Val x481 = (x479 - (x480 * arg0.bits[223]._super._super));
  Val x482 = (arg0.bits[97]._super._super + arg0.bits[224]._super._super);
  Val x483 = (arg0.bits[97]._super._super * Val(2));
  Val x484 = (x482 - (x483 * arg0.bits[224]._super._super));
  Val x485 = (arg0.bits[98]._super._super + arg0.bits[225]._super._super);
  Val x486 = (arg0.bits[98]._super._super * Val(2));
  Val x487 = (x485 - (x486 * arg0.bits[225]._super._super));
  Val x488 = (arg0.bits[99]._super._super + arg0.bits[226]._super._super);
  Val x489 = (arg0.bits[99]._super._super * Val(2));
  Val x490 = (x488 - (x489 * arg0.bits[226]._super._super));
  Val x491 = (arg0.bits[100]._super._super + arg0.bits[227]._super._super);
  Val x492 = (arg0.bits[100]._super._super * Val(2));
  Val x493 = (x491 - (x492 * arg0.bits[227]._super._super));
  Val x494 = (arg0.bits[101]._super._super + arg0.bits[228]._super._super);
  Val x495 = (arg0.bits[101]._super._super * Val(2));
  Val x496 = (x494 - (x495 * arg0.bits[228]._super._super));
  Val x497 = (arg0.bits[102]._super._super + arg0.bits[229]._super._super);
  Val x498 = (arg0.bits[102]._super._super * Val(2));
  Val x499 = (x497 - (x498 * arg0.bits[229]._super._super));
  Val x500 = (arg0.bits[103]._super._super + arg0.bits[230]._super._super);
  Val x501 = (arg0.bits[103]._super._super * Val(2));
  Val x502 = (x500 - (x501 * arg0.bits[230]._super._super));
  Val x503 = (arg0.bits[104]._super._super + arg0.bits[231]._super._super);
  Val x504 = (arg0.bits[104]._super._super * Val(2));
  Val x505 = (x503 - (x504 * arg0.bits[231]._super._super));
  Val x506 = (arg0.bits[105]._super._super + arg0.bits[232]._super._super);
  Val x507 = (arg0.bits[105]._super._super * Val(2));
  Val x508 = (x506 - (x507 * arg0.bits[232]._super._super));
  Val x509 = (arg0.bits[106]._super._super + arg0.bits[233]._super._super);
  Val x510 = (arg0.bits[106]._super._super * Val(2));
  Val x511 = (x509 - (x510 * arg0.bits[233]._super._super));
  Val x512 = (arg0.bits[107]._super._super + arg0.bits[234]._super._super);
  Val x513 = (arg0.bits[107]._super._super * Val(2));
  Val x514 = (x512 - (x513 * arg0.bits[234]._super._super));
  Val x515 = (arg0.bits[108]._super._super + arg0.bits[235]._super._super);
  Val x516 = (arg0.bits[108]._super._super * Val(2));
  Val x517 = (x515 - (x516 * arg0.bits[235]._super._super));
  Val x518 = (arg0.bits[109]._super._super + arg0.bits[236]._super._super);
  Val x519 = (arg0.bits[109]._super._super * Val(2));
  Val x520 = (x518 - (x519 * arg0.bits[236]._super._super));
  Val x521 = (arg0.bits[110]._super._super + arg0.bits[237]._super._super);
  Val x522 = (arg0.bits[110]._super._super * Val(2));
  Val x523 = (x521 - (x522 * arg0.bits[237]._super._super));
  Val x524 = (arg0.bits[111]._super._super + arg0.bits[238]._super._super);
  Val x525 = (arg0.bits[111]._super._super * Val(2));
  Val x526 = (x524 - (x525 * arg0.bits[238]._super._super));
  Val x527 = (arg0.bits[112]._super._super + arg0.bits[239]._super._super);
  Val x528 = (arg0.bits[112]._super._super * Val(2));
  Val x529 = (x527 - (x528 * arg0.bits[239]._super._super));
  Val x530 = (arg0.bits[113]._super._super + arg0.bits[240]._super._super);
  Val x531 = (arg0.bits[113]._super._super * Val(2));
  Val x532 = (x530 - (x531 * arg0.bits[240]._super._super));
  Val x533 = (arg0.bits[114]._super._super + arg0.bits[241]._super._super);
  Val x534 = (arg0.bits[114]._super._super * Val(2));
  Val x535 = (x533 - (x534 * arg0.bits[241]._super._super));
  Val x536 = (arg0.bits[115]._super._super + arg0.bits[242]._super._super);
  Val x537 = (arg0.bits[115]._super._super * Val(2));
  Val x538 = (x536 - (x537 * arg0.bits[242]._super._super));
  Val x539 = (arg0.bits[116]._super._super + arg0.bits[243]._super._super);
  Val x540 = (arg0.bits[116]._super._super * Val(2));
  Val x541 = (x539 - (x540 * arg0.bits[243]._super._super));
  Val x542 = (arg0.bits[117]._super._super + arg0.bits[244]._super._super);
  Val x543 = (arg0.bits[117]._super._super * Val(2));
  Val x544 = (x542 - (x543 * arg0.bits[244]._super._super));
  Val x545 = (arg0.bits[118]._super._super + arg0.bits[245]._super._super);
  Val x546 = (arg0.bits[118]._super._super * Val(2));
  Val x547 = (x545 - (x546 * arg0.bits[245]._super._super));
  Val x548 = (arg0.bits[119]._super._super + arg0.bits[246]._super._super);
  Val x549 = (arg0.bits[119]._super._super * Val(2));
  Val x550 = (x548 - (x549 * arg0.bits[246]._super._super));
  Val x551 = (arg0.bits[120]._super._super + arg0.bits[247]._super._super);
  Val x552 = (arg0.bits[120]._super._super * Val(2));
  Val x553 = (x551 - (x552 * arg0.bits[247]._super._super));
  Val x554 = (arg0.bits[121]._super._super + arg0.bits[248]._super._super);
  Val x555 = (arg0.bits[121]._super._super * Val(2));
  Val x556 = (x554 - (x555 * arg0.bits[248]._super._super));
  Val x557 = (arg0.bits[122]._super._super + arg0.bits[249]._super._super);
  Val x558 = (arg0.bits[122]._super._super * Val(2));
  Val x559 = (x557 - (x558 * arg0.bits[249]._super._super));
  Val x560 = (arg0.bits[123]._super._super + arg0.bits[250]._super._super);
  Val x561 = (arg0.bits[123]._super._super * Val(2));
  Val x562 = (x560 - (x561 * arg0.bits[250]._super._super));
  Val x563 = (arg0.bits[124]._super._super + arg0.bits[251]._super._super);
  Val x564 = (arg0.bits[124]._super._super * Val(2));
  Val x565 = (x563 - (x564 * arg0.bits[251]._super._super));
  Val x566 = (arg0.bits[125]._super._super + arg0.bits[252]._super._super);
  Val x567 = (arg0.bits[125]._super._super * Val(2));
  Val x568 = (x566 - (x567 * arg0.bits[252]._super._super));
  Val x569 = (arg0.bits[126]._super._super + arg0.bits[253]._super._super);
  Val x570 = (arg0.bits[126]._super._super * Val(2));
  Val x571 = (x569 - (x570 * arg0.bits[253]._super._super));
  Val x572 = (arg0.bits[127]._super._super + arg0.bits[254]._super._super);
  Val x573 = (arg0.bits[127]._super._super * Val(2));
  Val x574 = (x572 - (x573 * arg0.bits[254]._super._super));
  Val x575 = (arg0.bits[128]._super._super + arg0.bits[319]._super._super);
  Val x576 = (arg0.bits[128]._super._super * Val(2));
  Val x577 = (x575 - (x576 * arg0.bits[319]._super._super));
  Val x578 = (arg0.bits[129]._super._super + arg0.bits[256]._super._super);
  Val x579 = (arg0.bits[129]._super._super * Val(2));
  Val x580 = (x578 - (x579 * arg0.bits[256]._super._super));
  Val x581 = (arg0.bits[130]._super._super + arg0.bits[257]._super._super);
  Val x582 = (arg0.bits[130]._super._super * Val(2));
  Val x583 = (x581 - (x582 * arg0.bits[257]._super._super));
  Val x584 = (arg0.bits[131]._super._super + arg0.bits[258]._super._super);
  Val x585 = (arg0.bits[131]._super._super * Val(2));
  Val x586 = (x584 - (x585 * arg0.bits[258]._super._super));
  Val x587 = (arg0.bits[132]._super._super + arg0.bits[259]._super._super);
  Val x588 = (arg0.bits[132]._super._super * Val(2));
  Val x589 = (x587 - (x588 * arg0.bits[259]._super._super));
  Val x590 = (arg0.bits[133]._super._super + arg0.bits[260]._super._super);
  Val x591 = (arg0.bits[133]._super._super * Val(2));
  Val x592 = (x590 - (x591 * arg0.bits[260]._super._super));
  Val x593 = (arg0.bits[134]._super._super + arg0.bits[261]._super._super);
  Val x594 = (arg0.bits[134]._super._super * Val(2));
  Val x595 = (x593 - (x594 * arg0.bits[261]._super._super));
  Val x596 = (arg0.bits[135]._super._super + arg0.bits[262]._super._super);
  Val x597 = (arg0.bits[135]._super._super * Val(2));
  Val x598 = (x596 - (x597 * arg0.bits[262]._super._super));
  Val x599 = (arg0.bits[136]._super._super + arg0.bits[263]._super._super);
  Val x600 = (arg0.bits[136]._super._super * Val(2));
  Val x601 = (x599 - (x600 * arg0.bits[263]._super._super));
  Val x602 = (arg0.bits[137]._super._super + arg0.bits[264]._super._super);
  Val x603 = (arg0.bits[137]._super._super * Val(2));
  Val x604 = (x602 - (x603 * arg0.bits[264]._super._super));
  Val x605 = (arg0.bits[138]._super._super + arg0.bits[265]._super._super);
  Val x606 = (arg0.bits[138]._super._super * Val(2));
  Val x607 = (x605 - (x606 * arg0.bits[265]._super._super));
  Val x608 = (arg0.bits[139]._super._super + arg0.bits[266]._super._super);
  Val x609 = (arg0.bits[139]._super._super * Val(2));
  Val x610 = (x608 - (x609 * arg0.bits[266]._super._super));
  Val x611 = (arg0.bits[140]._super._super + arg0.bits[267]._super._super);
  Val x612 = (arg0.bits[140]._super._super * Val(2));
  Val x613 = (x611 - (x612 * arg0.bits[267]._super._super));
  Val x614 = (arg0.bits[141]._super._super + arg0.bits[268]._super._super);
  Val x615 = (arg0.bits[141]._super._super * Val(2));
  Val x616 = (x614 - (x615 * arg0.bits[268]._super._super));
  Val x617 = (arg0.bits[142]._super._super + arg0.bits[269]._super._super);
  Val x618 = (arg0.bits[142]._super._super * Val(2));
  Val x619 = (x617 - (x618 * arg0.bits[269]._super._super));
  Val x620 = (arg0.bits[143]._super._super + arg0.bits[270]._super._super);
  Val x621 = (arg0.bits[143]._super._super * Val(2));
  Val x622 = (x620 - (x621 * arg0.bits[270]._super._super));
  Val x623 = (arg0.bits[144]._super._super + arg0.bits[271]._super._super);
  Val x624 = (arg0.bits[144]._super._super * Val(2));
  Val x625 = (x623 - (x624 * arg0.bits[271]._super._super));
  Val x626 = (arg0.bits[145]._super._super + arg0.bits[272]._super._super);
  Val x627 = (arg0.bits[145]._super._super * Val(2));
  Val x628 = (x626 - (x627 * arg0.bits[272]._super._super));
  Val x629 = (arg0.bits[146]._super._super + arg0.bits[273]._super._super);
  Val x630 = (arg0.bits[146]._super._super * Val(2));
  Val x631 = (x629 - (x630 * arg0.bits[273]._super._super));
  Val x632 = (arg0.bits[147]._super._super + arg0.bits[274]._super._super);
  Val x633 = (arg0.bits[147]._super._super * Val(2));
  Val x634 = (x632 - (x633 * arg0.bits[274]._super._super));
  Val x635 = (arg0.bits[148]._super._super + arg0.bits[275]._super._super);
  Val x636 = (arg0.bits[148]._super._super * Val(2));
  Val x637 = (x635 - (x636 * arg0.bits[275]._super._super));
  Val x638 = (arg0.bits[149]._super._super + arg0.bits[276]._super._super);
  Val x639 = (arg0.bits[149]._super._super * Val(2));
  Val x640 = (x638 - (x639 * arg0.bits[276]._super._super));
  Val x641 = (arg0.bits[150]._super._super + arg0.bits[277]._super._super);
  Val x642 = (arg0.bits[150]._super._super * Val(2));
  Val x643 = (x641 - (x642 * arg0.bits[277]._super._super));
  Val x644 = (arg0.bits[151]._super._super + arg0.bits[278]._super._super);
  Val x645 = (arg0.bits[151]._super._super * Val(2));
  Val x646 = (x644 - (x645 * arg0.bits[278]._super._super));
  Val x647 = (arg0.bits[152]._super._super + arg0.bits[279]._super._super);
  Val x648 = (arg0.bits[152]._super._super * Val(2));
  Val x649 = (x647 - (x648 * arg0.bits[279]._super._super));
  Val x650 = (arg0.bits[153]._super._super + arg0.bits[280]._super._super);
  Val x651 = (arg0.bits[153]._super._super * Val(2));
  Val x652 = (x650 - (x651 * arg0.bits[280]._super._super));
  Val x653 = (arg0.bits[154]._super._super + arg0.bits[281]._super._super);
  Val x654 = (arg0.bits[154]._super._super * Val(2));
  Val x655 = (x653 - (x654 * arg0.bits[281]._super._super));
  Val x656 = (arg0.bits[155]._super._super + arg0.bits[282]._super._super);
  Val x657 = (arg0.bits[155]._super._super * Val(2));
  Val x658 = (x656 - (x657 * arg0.bits[282]._super._super));
  Val x659 = (arg0.bits[156]._super._super + arg0.bits[283]._super._super);
  Val x660 = (arg0.bits[156]._super._super * Val(2));
  Val x661 = (x659 - (x660 * arg0.bits[283]._super._super));
  Val x662 = (arg0.bits[157]._super._super + arg0.bits[284]._super._super);
  Val x663 = (arg0.bits[157]._super._super * Val(2));
  Val x664 = (x662 - (x663 * arg0.bits[284]._super._super));
  Val x665 = (arg0.bits[158]._super._super + arg0.bits[285]._super._super);
  Val x666 = (arg0.bits[158]._super._super * Val(2));
  Val x667 = (x665 - (x666 * arg0.bits[285]._super._super));
  Val x668 = (arg0.bits[159]._super._super + arg0.bits[286]._super._super);
  Val x669 = (arg0.bits[159]._super._super * Val(2));
  Val x670 = (x668 - (x669 * arg0.bits[286]._super._super));
  Val x671 = (arg0.bits[160]._super._super + arg0.bits[287]._super._super);
  Val x672 = (arg0.bits[160]._super._super * Val(2));
  Val x673 = (x671 - (x672 * arg0.bits[287]._super._super));
  Val x674 = (arg0.bits[161]._super._super + arg0.bits[288]._super._super);
  Val x675 = (arg0.bits[161]._super._super * Val(2));
  Val x676 = (x674 - (x675 * arg0.bits[288]._super._super));
  Val x677 = (arg0.bits[162]._super._super + arg0.bits[289]._super._super);
  Val x678 = (arg0.bits[162]._super._super * Val(2));
  Val x679 = (x677 - (x678 * arg0.bits[289]._super._super));
  Val x680 = (arg0.bits[163]._super._super + arg0.bits[290]._super._super);
  Val x681 = (arg0.bits[163]._super._super * Val(2));
  Val x682 = (x680 - (x681 * arg0.bits[290]._super._super));
  Val x683 = (arg0.bits[164]._super._super + arg0.bits[291]._super._super);
  Val x684 = (arg0.bits[164]._super._super * Val(2));
  Val x685 = (x683 - (x684 * arg0.bits[291]._super._super));
  Val x686 = (arg0.bits[165]._super._super + arg0.bits[292]._super._super);
  Val x687 = (arg0.bits[165]._super._super * Val(2));
  Val x688 = (x686 - (x687 * arg0.bits[292]._super._super));
  Val x689 = (arg0.bits[166]._super._super + arg0.bits[293]._super._super);
  Val x690 = (arg0.bits[166]._super._super * Val(2));
  Val x691 = (x689 - (x690 * arg0.bits[293]._super._super));
  Val x692 = (arg0.bits[167]._super._super + arg0.bits[294]._super._super);
  Val x693 = (arg0.bits[167]._super._super * Val(2));
  Val x694 = (x692 - (x693 * arg0.bits[294]._super._super));
  Val x695 = (arg0.bits[168]._super._super + arg0.bits[295]._super._super);
  Val x696 = (arg0.bits[168]._super._super * Val(2));
  Val x697 = (x695 - (x696 * arg0.bits[295]._super._super));
  Val x698 = (arg0.bits[169]._super._super + arg0.bits[296]._super._super);
  Val x699 = (arg0.bits[169]._super._super * Val(2));
  Val x700 = (x698 - (x699 * arg0.bits[296]._super._super));
  Val x701 = (arg0.bits[170]._super._super + arg0.bits[297]._super._super);
  Val x702 = (arg0.bits[170]._super._super * Val(2));
  Val x703 = (x701 - (x702 * arg0.bits[297]._super._super));
  Val x704 = (arg0.bits[171]._super._super + arg0.bits[298]._super._super);
  Val x705 = (arg0.bits[171]._super._super * Val(2));
  Val x706 = (x704 - (x705 * arg0.bits[298]._super._super));
  Val x707 = (arg0.bits[172]._super._super + arg0.bits[299]._super._super);
  Val x708 = (arg0.bits[172]._super._super * Val(2));
  Val x709 = (x707 - (x708 * arg0.bits[299]._super._super));
  Val x710 = (arg0.bits[173]._super._super + arg0.bits[300]._super._super);
  Val x711 = (arg0.bits[173]._super._super * Val(2));
  Val x712 = (x710 - (x711 * arg0.bits[300]._super._super));
  Val x713 = (arg0.bits[174]._super._super + arg0.bits[301]._super._super);
  Val x714 = (arg0.bits[174]._super._super * Val(2));
  Val x715 = (x713 - (x714 * arg0.bits[301]._super._super));
  Val x716 = (arg0.bits[175]._super._super + arg0.bits[302]._super._super);
  Val x717 = (arg0.bits[175]._super._super * Val(2));
  Val x718 = (x716 - (x717 * arg0.bits[302]._super._super));
  Val x719 = (arg0.bits[176]._super._super + arg0.bits[303]._super._super);
  Val x720 = (arg0.bits[176]._super._super * Val(2));
  Val x721 = (x719 - (x720 * arg0.bits[303]._super._super));
  Val x722 = (arg0.bits[177]._super._super + arg0.bits[304]._super._super);
  Val x723 = (arg0.bits[177]._super._super * Val(2));
  Val x724 = (x722 - (x723 * arg0.bits[304]._super._super));
  Val x725 = (arg0.bits[178]._super._super + arg0.bits[305]._super._super);
  Val x726 = (arg0.bits[178]._super._super * Val(2));
  Val x727 = (x725 - (x726 * arg0.bits[305]._super._super));
  Val x728 = (arg0.bits[179]._super._super + arg0.bits[306]._super._super);
  Val x729 = (arg0.bits[179]._super._super * Val(2));
  Val x730 = (x728 - (x729 * arg0.bits[306]._super._super));
  Val x731 = (arg0.bits[180]._super._super + arg0.bits[307]._super._super);
  Val x732 = (arg0.bits[180]._super._super * Val(2));
  Val x733 = (x731 - (x732 * arg0.bits[307]._super._super));
  Val x734 = (arg0.bits[181]._super._super + arg0.bits[308]._super._super);
  Val x735 = (arg0.bits[181]._super._super * Val(2));
  Val x736 = (x734 - (x735 * arg0.bits[308]._super._super));
  Val x737 = (arg0.bits[182]._super._super + arg0.bits[309]._super._super);
  Val x738 = (arg0.bits[182]._super._super * Val(2));
  Val x739 = (x737 - (x738 * arg0.bits[309]._super._super));
  Val x740 = (arg0.bits[183]._super._super + arg0.bits[310]._super._super);
  Val x741 = (arg0.bits[183]._super._super * Val(2));
  Val x742 = (x740 - (x741 * arg0.bits[310]._super._super));
  Val x743 = (arg0.bits[184]._super._super + arg0.bits[311]._super._super);
  Val x744 = (arg0.bits[184]._super._super * Val(2));
  Val x745 = (x743 - (x744 * arg0.bits[311]._super._super));
  Val x746 = (arg0.bits[185]._super._super + arg0.bits[312]._super._super);
  Val x747 = (arg0.bits[185]._super._super * Val(2));
  Val x748 = (x746 - (x747 * arg0.bits[312]._super._super));
  Val x749 = (arg0.bits[186]._super._super + arg0.bits[313]._super._super);
  Val x750 = (arg0.bits[186]._super._super * Val(2));
  Val x751 = (x749 - (x750 * arg0.bits[313]._super._super));
  Val x752 = (arg0.bits[187]._super._super + arg0.bits[314]._super._super);
  Val x753 = (arg0.bits[187]._super._super * Val(2));
  Val x754 = (x752 - (x753 * arg0.bits[314]._super._super));
  Val x755 = (arg0.bits[188]._super._super + arg0.bits[315]._super._super);
  Val x756 = (arg0.bits[188]._super._super * Val(2));
  Val x757 = (x755 - (x756 * arg0.bits[315]._super._super));
  Val x758 = (arg0.bits[189]._super._super + arg0.bits[316]._super._super);
  Val x759 = (arg0.bits[189]._super._super * Val(2));
  Val x760 = (x758 - (x759 * arg0.bits[316]._super._super));
  Val x761 = (arg0.bits[190]._super._super + arg0.bits[317]._super._super);
  Val x762 = (arg0.bits[190]._super._super * Val(2));
  Val x763 = (x761 - (x762 * arg0.bits[317]._super._super));
  Val x764 = (arg0.bits[191]._super._super + arg0.bits[318]._super._super);
  Val x765 = (arg0.bits[191]._super._super * Val(2));
  Val x766 = (x764 - (x765 * arg0.bits[318]._super._super));
  Val x767 = (arg0.bits[192]._super._super + arg0.bits[63]._super._super);
  Val x768 = (arg0.bits[192]._super._super * Val(2));
  Val x769 = (x767 - (x768 * arg0.bits[63]._super._super));
  Val x770 = (arg0.bits[193]._super._super + arg0.bits[0]._super._super);
  Val x771 = (arg0.bits[193]._super._super * Val(2));
  Val x772 = (x770 - (x771 * arg0.bits[0]._super._super));
  Val x773 = (arg0.bits[194]._super._super + arg0.bits[1]._super._super);
  Val x774 = (arg0.bits[194]._super._super * Val(2));
  Val x775 = (x773 - (x774 * arg0.bits[1]._super._super));
  Val x776 = (arg0.bits[195]._super._super + arg0.bits[2]._super._super);
  Val x777 = (arg0.bits[195]._super._super * Val(2));
  Val x778 = (x776 - (x777 * arg0.bits[2]._super._super));
  Val x779 = (arg0.bits[196]._super._super + arg0.bits[3]._super._super);
  Val x780 = (arg0.bits[196]._super._super * Val(2));
  Val x781 = (x779 - (x780 * arg0.bits[3]._super._super));
  Val x782 = (arg0.bits[197]._super._super + arg0.bits[4]._super._super);
  Val x783 = (arg0.bits[197]._super._super * Val(2));
  Val x784 = (x782 - (x783 * arg0.bits[4]._super._super));
  Val x785 = (arg0.bits[198]._super._super + arg0.bits[5]._super._super);
  Val x786 = (arg0.bits[198]._super._super * Val(2));
  Val x787 = (x785 - (x786 * arg0.bits[5]._super._super));
  Val x788 = (arg0.bits[199]._super._super + arg0.bits[6]._super._super);
  Val x789 = (arg0.bits[199]._super._super * Val(2));
  Val x790 = (x788 - (x789 * arg0.bits[6]._super._super));
  Val x791 = (arg0.bits[200]._super._super + arg0.bits[7]._super._super);
  Val x792 = (arg0.bits[200]._super._super * Val(2));
  Val x793 = (x791 - (x792 * arg0.bits[7]._super._super));
  Val x794 = (arg0.bits[201]._super._super + arg0.bits[8]._super._super);
  Val x795 = (arg0.bits[201]._super._super * Val(2));
  Val x796 = (x794 - (x795 * arg0.bits[8]._super._super));
  Val x797 = (arg0.bits[202]._super._super + arg0.bits[9]._super._super);
  Val x798 = (arg0.bits[202]._super._super * Val(2));
  Val x799 = (x797 - (x798 * arg0.bits[9]._super._super));
  Val x800 = (arg0.bits[203]._super._super + arg0.bits[10]._super._super);
  Val x801 = (arg0.bits[203]._super._super * Val(2));
  Val x802 = (x800 - (x801 * arg0.bits[10]._super._super));
  Val x803 = (arg0.bits[204]._super._super + arg0.bits[11]._super._super);
  Val x804 = (arg0.bits[204]._super._super * Val(2));
  Val x805 = (x803 - (x804 * arg0.bits[11]._super._super));
  Val x806 = (arg0.bits[205]._super._super + arg0.bits[12]._super._super);
  Val x807 = (arg0.bits[205]._super._super * Val(2));
  Val x808 = (x806 - (x807 * arg0.bits[12]._super._super));
  Val x809 = (arg0.bits[206]._super._super + arg0.bits[13]._super._super);
  Val x810 = (arg0.bits[206]._super._super * Val(2));
  Val x811 = (x809 - (x810 * arg0.bits[13]._super._super));
  Val x812 = (arg0.bits[207]._super._super + arg0.bits[14]._super._super);
  Val x813 = (arg0.bits[207]._super._super * Val(2));
  Val x814 = (x812 - (x813 * arg0.bits[14]._super._super));
  Val x815 = (arg0.bits[208]._super._super + arg0.bits[15]._super._super);
  Val x816 = (arg0.bits[208]._super._super * Val(2));
  Val x817 = (x815 - (x816 * arg0.bits[15]._super._super));
  Val x818 = (arg0.bits[209]._super._super + arg0.bits[16]._super._super);
  Val x819 = (arg0.bits[209]._super._super * Val(2));
  Val x820 = (x818 - (x819 * arg0.bits[16]._super._super));
  Val x821 = (arg0.bits[210]._super._super + arg0.bits[17]._super._super);
  Val x822 = (arg0.bits[210]._super._super * Val(2));
  Val x823 = (x821 - (x822 * arg0.bits[17]._super._super));
  Val x824 = (arg0.bits[211]._super._super + arg0.bits[18]._super._super);
  Val x825 = (arg0.bits[211]._super._super * Val(2));
  Val x826 = (x824 - (x825 * arg0.bits[18]._super._super));
  Val x827 = (arg0.bits[212]._super._super + arg0.bits[19]._super._super);
  Val x828 = (arg0.bits[212]._super._super * Val(2));
  Val x829 = (x827 - (x828 * arg0.bits[19]._super._super));
  Val x830 = (arg0.bits[213]._super._super + arg0.bits[20]._super._super);
  Val x831 = (arg0.bits[213]._super._super * Val(2));
  Val x832 = (x830 - (x831 * arg0.bits[20]._super._super));
  Val x833 = (arg0.bits[214]._super._super + arg0.bits[21]._super._super);
  Val x834 = (arg0.bits[214]._super._super * Val(2));
  Val x835 = (x833 - (x834 * arg0.bits[21]._super._super));
  Val x836 = (arg0.bits[215]._super._super + arg0.bits[22]._super._super);
  Val x837 = (arg0.bits[215]._super._super * Val(2));
  Val x838 = (x836 - (x837 * arg0.bits[22]._super._super));
  Val x839 = (arg0.bits[216]._super._super + arg0.bits[23]._super._super);
  Val x840 = (arg0.bits[216]._super._super * Val(2));
  Val x841 = (x839 - (x840 * arg0.bits[23]._super._super));
  Val x842 = (arg0.bits[217]._super._super + arg0.bits[24]._super._super);
  Val x843 = (arg0.bits[217]._super._super * Val(2));
  Val x844 = (x842 - (x843 * arg0.bits[24]._super._super));
  Val x845 = (arg0.bits[218]._super._super + arg0.bits[25]._super._super);
  Val x846 = (arg0.bits[218]._super._super * Val(2));
  Val x847 = (x845 - (x846 * arg0.bits[25]._super._super));
  Val x848 = (arg0.bits[219]._super._super + arg0.bits[26]._super._super);
  Val x849 = (arg0.bits[219]._super._super * Val(2));
  Val x850 = (x848 - (x849 * arg0.bits[26]._super._super));
  Val x851 = (arg0.bits[220]._super._super + arg0.bits[27]._super._super);
  Val x852 = (arg0.bits[220]._super._super * Val(2));
  Val x853 = (x851 - (x852 * arg0.bits[27]._super._super));
  Val x854 = (arg0.bits[221]._super._super + arg0.bits[28]._super._super);
  Val x855 = (arg0.bits[221]._super._super * Val(2));
  Val x856 = (x854 - (x855 * arg0.bits[28]._super._super));
  Val x857 = (arg0.bits[222]._super._super + arg0.bits[29]._super._super);
  Val x858 = (arg0.bits[222]._super._super * Val(2));
  Val x859 = (x857 - (x858 * arg0.bits[29]._super._super));
  Val x860 = (arg0.bits[223]._super._super + arg0.bits[30]._super._super);
  Val x861 = (arg0.bits[223]._super._super * Val(2));
  Val x862 = (x860 - (x861 * arg0.bits[30]._super._super));
  Val x863 = (arg0.bits[224]._super._super + arg0.bits[31]._super._super);
  Val x864 = (arg0.bits[224]._super._super * Val(2));
  Val x865 = (x863 - (x864 * arg0.bits[31]._super._super));
  Val x866 = (arg0.bits[225]._super._super + arg0.bits[32]._super._super);
  Val x867 = (arg0.bits[225]._super._super * Val(2));
  Val x868 = (x866 - (x867 * arg0.bits[32]._super._super));
  Val x869 = (arg0.bits[226]._super._super + arg0.bits[33]._super._super);
  Val x870 = (arg0.bits[226]._super._super * Val(2));
  Val x871 = (x869 - (x870 * arg0.bits[33]._super._super));
  Val x872 = (arg0.bits[227]._super._super + arg0.bits[34]._super._super);
  Val x873 = (arg0.bits[227]._super._super * Val(2));
  Val x874 = (x872 - (x873 * arg0.bits[34]._super._super));
  Val x875 = (arg0.bits[228]._super._super + arg0.bits[35]._super._super);
  Val x876 = (arg0.bits[228]._super._super * Val(2));
  Val x877 = (x875 - (x876 * arg0.bits[35]._super._super));
  Val x878 = (arg0.bits[229]._super._super + arg0.bits[36]._super._super);
  Val x879 = (arg0.bits[229]._super._super * Val(2));
  Val x880 = (x878 - (x879 * arg0.bits[36]._super._super));
  Val x881 = (arg0.bits[230]._super._super + arg0.bits[37]._super._super);
  Val x882 = (arg0.bits[230]._super._super * Val(2));
  Val x883 = (x881 - (x882 * arg0.bits[37]._super._super));
  Val x884 = (arg0.bits[231]._super._super + arg0.bits[38]._super._super);
  Val x885 = (arg0.bits[231]._super._super * Val(2));
  Val x886 = (x884 - (x885 * arg0.bits[38]._super._super));
  Val x887 = (arg0.bits[232]._super._super + arg0.bits[39]._super._super);
  Val x888 = (arg0.bits[232]._super._super * Val(2));
  Val x889 = (x887 - (x888 * arg0.bits[39]._super._super));
  Val x890 = (arg0.bits[233]._super._super + arg0.bits[40]._super._super);
  Val x891 = (arg0.bits[233]._super._super * Val(2));
  Val x892 = (x890 - (x891 * arg0.bits[40]._super._super));
  Val x893 = (arg0.bits[234]._super._super + arg0.bits[41]._super._super);
  Val x894 = (arg0.bits[234]._super._super * Val(2));
  Val x895 = (x893 - (x894 * arg0.bits[41]._super._super));
  Val x896 = (arg0.bits[235]._super._super + arg0.bits[42]._super._super);
  Val x897 = (arg0.bits[235]._super._super * Val(2));
  Val x898 = (x896 - (x897 * arg0.bits[42]._super._super));
  Val x899 = (arg0.bits[236]._super._super + arg0.bits[43]._super._super);
  Val x900 = (arg0.bits[236]._super._super * Val(2));
  Val x901 = (x899 - (x900 * arg0.bits[43]._super._super));
  Val x902 = (arg0.bits[237]._super._super + arg0.bits[44]._super._super);
  Val x903 = (arg0.bits[237]._super._super * Val(2));
  Val x904 = (x902 - (x903 * arg0.bits[44]._super._super));
  Val x905 = (arg0.bits[238]._super._super + arg0.bits[45]._super._super);
  Val x906 = (arg0.bits[238]._super._super * Val(2));
  Val x907 = (x905 - (x906 * arg0.bits[45]._super._super));
  Val x908 = (arg0.bits[239]._super._super + arg0.bits[46]._super._super);
  Val x909 = (arg0.bits[239]._super._super * Val(2));
  Val x910 = (x908 - (x909 * arg0.bits[46]._super._super));
  Val x911 = (arg0.bits[240]._super._super + arg0.bits[47]._super._super);
  Val x912 = (arg0.bits[240]._super._super * Val(2));
  Val x913 = (x911 - (x912 * arg0.bits[47]._super._super));
  Val x914 = (arg0.bits[241]._super._super + arg0.bits[48]._super._super);
  Val x915 = (arg0.bits[241]._super._super * Val(2));
  Val x916 = (x914 - (x915 * arg0.bits[48]._super._super));
  Val x917 = (arg0.bits[242]._super._super + arg0.bits[49]._super._super);
  Val x918 = (arg0.bits[242]._super._super * Val(2));
  Val x919 = (x917 - (x918 * arg0.bits[49]._super._super));
  Val x920 = (arg0.bits[243]._super._super + arg0.bits[50]._super._super);
  Val x921 = (arg0.bits[243]._super._super * Val(2));
  Val x922 = (x920 - (x921 * arg0.bits[50]._super._super));
  Val x923 = (arg0.bits[244]._super._super + arg0.bits[51]._super._super);
  Val x924 = (arg0.bits[244]._super._super * Val(2));
  Val x925 = (x923 - (x924 * arg0.bits[51]._super._super));
  Val x926 = (arg0.bits[245]._super._super + arg0.bits[52]._super._super);
  Val x927 = (arg0.bits[245]._super._super * Val(2));
  Val x928 = (x926 - (x927 * arg0.bits[52]._super._super));
  Val x929 = (arg0.bits[246]._super._super + arg0.bits[53]._super._super);
  Val x930 = (arg0.bits[246]._super._super * Val(2));
  Val x931 = (x929 - (x930 * arg0.bits[53]._super._super));
  Val x932 = (arg0.bits[247]._super._super + arg0.bits[54]._super._super);
  Val x933 = (arg0.bits[247]._super._super * Val(2));
  Val x934 = (x932 - (x933 * arg0.bits[54]._super._super));
  Val x935 = (arg0.bits[249]._super._super + arg0.bits[56]._super._super);
  Val x936 = (arg0.bits[249]._super._super * Val(2));
  Val x937 = (x935 - (x936 * arg0.bits[56]._super._super));
  Val x938 = (arg0.bits[250]._super._super + arg0.bits[57]._super._super);
  Val x939 = (arg0.bits[250]._super._super * Val(2));
  Val x940 = (x938 - (x939 * arg0.bits[57]._super._super));
  Val x941 = (arg0.bits[251]._super._super + arg0.bits[58]._super._super);
  Val x942 = (arg0.bits[251]._super._super * Val(2));
  Val x943 = (x941 - (x942 * arg0.bits[58]._super._super));
  Val x944 = (arg0.bits[252]._super._super + arg0.bits[59]._super._super);
  Val x945 = (arg0.bits[252]._super._super * Val(2));
  Val x946 = (x944 - (x945 * arg0.bits[59]._super._super));
  Val x947 = (arg0.bits[253]._super._super + arg0.bits[60]._super._super);
  Val x948 = (arg0.bits[253]._super._super * Val(2));
  Val x949 = (x947 - (x948 * arg0.bits[60]._super._super));
  Val x950 = (arg0.bits[254]._super._super + arg0.bits[61]._super._super);
  Val x951 = (arg0.bits[254]._super._super * Val(2));
  Val x952 = (x950 - (x951 * arg0.bits[61]._super._super));
  Val x953 = (arg0.bits[255]._super._super + arg0.bits[62]._super._super);
  Val x954 = (arg0.bits[255]._super._super * Val(2));
  Val x955 = (x953 - (x954 * arg0.bits[62]._super._super));
  // ThetaP2(zirgen/circuit/keccak2/keccak.zir:31)
  Val x956 = (arg2.bits[0]._super._super + x103);
  Val x957 = (arg2.bits[0]._super._super * Val(2));
  Val x958 = (arg2.bits[1]._super._super + x106);
  Val x959 = (arg2.bits[1]._super._super * Val(2));
  Val x960 = (arg2.bits[2]._super._super + x109);
  Val x961 = (arg2.bits[2]._super._super * Val(2));
  Val x962 = (arg2.bits[3]._super._super + x112);
  Val x963 = (arg2.bits[3]._super._super * Val(2));
  Val x964 = (arg2.bits[4]._super._super + x115);
  Val x965 = (arg2.bits[4]._super._super * Val(2));
  Val x966 = (arg2.bits[5]._super._super + x118);
  Val x967 = (arg2.bits[5]._super._super * Val(2));
  Val x968 = (arg2.bits[6]._super._super + x121);
  Val x969 = (arg2.bits[6]._super._super * Val(2));
  Val x970 = (arg2.bits[7]._super._super + x124);
  Val x971 = (arg2.bits[7]._super._super * Val(2));
  Val x972 = (arg2.bits[8]._super._super + x127);
  Val x973 = (arg2.bits[8]._super._super * Val(2));
  Val x974 = (arg2.bits[9]._super._super + x130);
  Val x975 = (arg2.bits[9]._super._super * Val(2));
  Val x976 = (arg2.bits[10]._super._super + x133);
  Val x977 = (arg2.bits[10]._super._super * Val(2));
  Val x978 = (arg2.bits[11]._super._super + x136);
  Val x979 = (arg2.bits[11]._super._super * Val(2));
  Val x980 = (arg2.bits[12]._super._super + x139);
  Val x981 = (arg2.bits[12]._super._super * Val(2));
  Val x982 = (arg2.bits[13]._super._super + x142);
  Val x983 = (arg2.bits[13]._super._super * Val(2));
  Val x984 = (arg2.bits[14]._super._super + x145);
  Val x985 = (arg2.bits[14]._super._super * Val(2));
  Val x986 = (arg2.bits[15]._super._super + x148);
  Val x987 = (arg2.bits[15]._super._super * Val(2));
  Val x988 = (arg2.bits[16]._super._super + x151);
  Val x989 = (arg2.bits[16]._super._super * Val(2));
  Val x990 = (arg2.bits[17]._super._super + x154);
  Val x991 = (arg2.bits[17]._super._super * Val(2));
  Val x992 = (arg2.bits[18]._super._super + x157);
  Val x993 = (arg2.bits[18]._super._super * Val(2));
  Val x994 = (arg2.bits[19]._super._super + x160);
  Val x995 = (arg2.bits[19]._super._super * Val(2));
  Val x996 = (arg2.bits[20]._super._super + x163);
  Val x997 = (arg2.bits[20]._super._super * Val(2));
  Val x998 = (arg2.bits[21]._super._super + x166);
  Val x999 = (arg2.bits[21]._super._super * Val(2));
  Val x1000 = (arg2.bits[22]._super._super + x169);
  Val x1001 = (arg2.bits[22]._super._super * Val(2));
  Val x1002 = (arg2.bits[23]._super._super + x172);
  Val x1003 = (arg2.bits[23]._super._super * Val(2));
  Val x1004 = (arg2.bits[24]._super._super + x175);
  Val x1005 = (arg2.bits[24]._super._super * Val(2));
  Val x1006 = (arg2.bits[25]._super._super + x178);
  Val x1007 = (arg2.bits[25]._super._super * Val(2));
  Val x1008 = (arg2.bits[26]._super._super + x181);
  Val x1009 = (arg2.bits[26]._super._super * Val(2));
  Val x1010 = (arg2.bits[27]._super._super + x184);
  Val x1011 = (arg2.bits[27]._super._super * Val(2));
  Val x1012 = (arg2.bits[28]._super._super + x187);
  Val x1013 = (arg2.bits[28]._super._super * Val(2));
  Val x1014 = (arg2.bits[29]._super._super + x190);
  Val x1015 = (arg2.bits[29]._super._super * Val(2));
  Val x1016 = (arg2.bits[30]._super._super + x193);
  Val x1017 = (arg2.bits[30]._super._super * Val(2));
  Val x1018 = (arg2.bits[31]._super._super + x196);
  Val x1019 = (arg2.bits[31]._super._super * Val(2));
  Val x1020 = (arg1.bits[63]._super._super + x286);
  Val x1021 = (arg1.bits[63]._super._super * Val(2));
  Val x1022 = (arg2.bits[32]._super._super + x289);
  Val x1023 = (arg2.bits[32]._super._super * Val(2));
  Val x1024 = (arg2.bits[33]._super._super + x292);
  Val x1025 = (arg2.bits[33]._super._super * Val(2));
  Val x1026 = (arg2.bits[34]._super._super + x295);
  Val x1027 = (arg2.bits[34]._super._super * Val(2));
  Val x1028 = (arg2.bits[35]._super._super + x298);
  Val x1029 = (arg2.bits[35]._super._super * Val(2));
  Val x1030 = (arg2.bits[36]._super._super + x301);
  Val x1031 = (arg2.bits[36]._super._super * Val(2));
  Val x1032 = (arg2.bits[37]._super._super + x304);
  Val x1033 = (arg2.bits[37]._super._super * Val(2));
  Val x1034 = (arg2.bits[38]._super._super + x307);
  Val x1035 = (arg2.bits[38]._super._super * Val(2));
  Val x1036 = (arg2.bits[39]._super._super + x310);
  Val x1037 = (arg2.bits[39]._super._super * Val(2));
  Val x1038 = (arg2.bits[40]._super._super + x313);
  Val x1039 = (arg2.bits[40]._super._super * Val(2));
  Val x1040 = (arg2.bits[41]._super._super + x316);
  Val x1041 = (arg2.bits[41]._super._super * Val(2));
  Val x1042 = (arg2.bits[42]._super._super + x319);
  Val x1043 = (arg2.bits[42]._super._super * Val(2));
  Val x1044 = (arg2.bits[43]._super._super + x322);
  Val x1045 = (arg2.bits[43]._super._super * Val(2));
  Val x1046 = (arg2.bits[44]._super._super + x325);
  Val x1047 = (arg2.bits[44]._super._super * Val(2));
  Val x1048 = (arg2.bits[45]._super._super + x328);
  Val x1049 = (arg2.bits[45]._super._super * Val(2));
  Val x1050 = (arg2.bits[46]._super._super + x331);
  Val x1051 = (arg2.bits[46]._super._super * Val(2));
  Val x1052 = (arg2.bits[47]._super._super + x334);
  Val x1053 = (arg2.bits[47]._super._super * Val(2));
  Val x1054 = (arg2.bits[48]._super._super + x337);
  Val x1055 = (arg2.bits[48]._super._super * Val(2));
  Val x1056 = (arg2.bits[49]._super._super + x340);
  Val x1057 = (arg2.bits[49]._super._super * Val(2));
  Val x1058 = (arg2.bits[50]._super._super + x343);
  Val x1059 = (arg2.bits[50]._super._super * Val(2));
  Val x1060 = (arg2.bits[51]._super._super + x346);
  Val x1061 = (arg2.bits[51]._super._super * Val(2));
  Val x1062 = (arg2.bits[52]._super._super + x349);
  Val x1063 = (arg2.bits[52]._super._super * Val(2));
  Val x1064 = (arg2.bits[53]._super._super + x352);
  Val x1065 = (arg2.bits[53]._super._super * Val(2));
  Val x1066 = (arg2.bits[54]._super._super + x355);
  Val x1067 = (arg2.bits[54]._super._super * Val(2));
  Val x1068 = (arg2.bits[55]._super._super + x358);
  Val x1069 = (arg2.bits[55]._super._super * Val(2));
  Val x1070 = (arg2.bits[56]._super._super + x361);
  Val x1071 = (arg2.bits[56]._super._super * Val(2));
  Val x1072 = (arg2.bits[57]._super._super + x364);
  Val x1073 = (arg2.bits[57]._super._super * Val(2));
  Val x1074 = (arg2.bits[58]._super._super + x367);
  Val x1075 = (arg2.bits[58]._super._super * Val(2));
  Val x1076 = (arg2.bits[59]._super._super + x370);
  Val x1077 = (arg2.bits[59]._super._super * Val(2));
  Val x1078 = (arg2.bits[60]._super._super + x373);
  Val x1079 = (arg2.bits[60]._super._super * Val(2));
  Val x1080 = (arg2.bits[61]._super._super + x376);
  Val x1081 = (arg2.bits[61]._super._super * Val(2));
  Val x1082 = (arg2.bits[62]._super._super + x379);
  Val x1083 = (arg2.bits[62]._super._super * Val(2));
  Val x1084 = (arg1.bits[64]._super._super + x385);
  Val x1085 = (arg1.bits[64]._super._super * Val(2));
  Val x1086 = (arg1.bits[65]._super._super + x388);
  Val x1087 = (arg1.bits[65]._super._super * Val(2));
  Val x1088 = (arg2.bits[66]._super._super + x487);
  Val x1089 = (arg2.bits[66]._super._super * Val(2));
  Val x1090 = (arg2.bits[67]._super._super + x490);
  Val x1091 = (arg2.bits[67]._super._super * Val(2));
  Val x1092 = (arg2.bits[68]._super._super + x493);
  Val x1093 = (arg2.bits[68]._super._super * Val(2));
  Val x1094 = (arg2.bits[69]._super._super + x496);
  Val x1095 = (arg2.bits[69]._super._super * Val(2));
  Val x1096 = (arg2.bits[70]._super._super + x499);
  Val x1097 = (arg2.bits[70]._super._super * Val(2));
  Val x1098 = (arg2.bits[71]._super._super + x502);
  Val x1099 = (arg2.bits[71]._super._super * Val(2));
  Val x1100 = (arg2.bits[72]._super._super + x505);
  Val x1101 = (arg2.bits[72]._super._super * Val(2));
  Val x1102 = (arg2.bits[73]._super._super + x508);
  Val x1103 = (arg2.bits[73]._super._super * Val(2));
  Val x1104 = (arg2.bits[74]._super._super + x511);
  Val x1105 = (arg2.bits[74]._super._super * Val(2));
  Val x1106 = (arg2.bits[75]._super._super + x514);
  Val x1107 = (arg2.bits[75]._super._super * Val(2));
  Val x1108 = (arg2.bits[76]._super._super + x517);
  Val x1109 = (arg2.bits[76]._super._super * Val(2));
  Val x1110 = (arg2.bits[77]._super._super + x520);
  Val x1111 = (arg2.bits[77]._super._super * Val(2));
  Val x1112 = (arg2.bits[78]._super._super + x523);
  Val x1113 = (arg2.bits[78]._super._super * Val(2));
  Val x1114 = (arg2.bits[79]._super._super + x526);
  Val x1115 = (arg2.bits[79]._super._super * Val(2));
  Val x1116 = (arg2.bits[80]._super._super + x529);
  Val x1117 = (arg2.bits[80]._super._super * Val(2));
  Val x1118 = (arg2.bits[81]._super._super + x532);
  Val x1119 = (arg2.bits[81]._super._super * Val(2));
  Val x1120 = (arg2.bits[82]._super._super + x535);
  Val x1121 = (arg2.bits[82]._super._super * Val(2));
  Val x1122 = (arg2.bits[83]._super._super + x538);
  Val x1123 = (arg2.bits[83]._super._super * Val(2));
  Val x1124 = (arg2.bits[84]._super._super + x541);
  Val x1125 = (arg2.bits[84]._super._super * Val(2));
  Val x1126 = (arg2.bits[85]._super._super + x544);
  Val x1127 = (arg2.bits[85]._super._super * Val(2));
  Val x1128 = (arg2.bits[86]._super._super + x547);
  Val x1129 = (arg2.bits[86]._super._super * Val(2));
  Val x1130 = (arg2.bits[87]._super._super + x550);
  Val x1131 = (arg2.bits[87]._super._super * Val(2));
  Val x1132 = (arg2.bits[88]._super._super + x553);
  Val x1133 = (arg2.bits[88]._super._super * Val(2));
  Val x1134 = (arg2.bits[89]._super._super + x556);
  Val x1135 = (arg2.bits[89]._super._super * Val(2));
  Val x1136 = (arg2.bits[90]._super._super + x559);
  Val x1137 = (arg2.bits[90]._super._super * Val(2));
  Val x1138 = (arg2.bits[91]._super._super + x562);
  Val x1139 = (arg2.bits[91]._super._super * Val(2));
  Val x1140 = (arg2.bits[92]._super._super + x565);
  Val x1141 = (arg2.bits[92]._super._super * Val(2));
  Val x1142 = (arg2.bits[93]._super._super + x568);
  Val x1143 = (arg2.bits[93]._super._super * Val(2));
  Val x1144 = (arg2.bits[94]._super._super + x571);
  Val x1145 = (arg2.bits[94]._super._super * Val(2));
  Val x1146 = (arg2.bits[95]._super._super + x574);
  Val x1147 = (arg2.bits[95]._super._super * Val(2));
  Val x1148 = (arg1.bits[100]._super._super + x589);
  Val x1149 = (arg1.bits[100]._super._super * Val(2));
  Val x1150 = (arg1.bits[101]._super._super + x592);
  Val x1151 = (arg1.bits[101]._super._super * Val(2));
  Val x1152 = (arg1.bits[102]._super._super + x595);
  Val x1153 = (arg1.bits[102]._super._super * Val(2));
  Val x1154 = (arg1.bits[103]._super._super + x598);
  Val x1155 = (arg1.bits[103]._super._super * Val(2));
  Val x1156 = (arg1.bits[104]._super._super + x601);
  Val x1157 = (arg1.bits[104]._super._super * Val(2));
  Val x1158 = (arg1.bits[105]._super._super + x604);
  Val x1159 = (arg1.bits[105]._super._super * Val(2));
  Val x1160 = (arg1.bits[106]._super._super + x607);
  Val x1161 = (arg1.bits[106]._super._super * Val(2));
  Val x1162 = (arg1.bits[107]._super._super + x610);
  Val x1163 = (arg1.bits[107]._super._super * Val(2));
  Val x1164 = (arg1.bits[108]._super._super + x613);
  Val x1165 = (arg1.bits[108]._super._super * Val(2));
  Val x1166 = (arg1.bits[109]._super._super + x616);
  Val x1167 = (arg1.bits[109]._super._super * Val(2));
  Val x1168 = (arg1.bits[110]._super._super + x619);
  Val x1169 = (arg1.bits[110]._super._super * Val(2));
  Val x1170 = (arg1.bits[111]._super._super + x622);
  Val x1171 = (arg1.bits[111]._super._super * Val(2));
  Val x1172 = (arg1.bits[112]._super._super + x625);
  Val x1173 = (arg1.bits[112]._super._super * Val(2));
  Val x1174 = (arg1.bits[113]._super._super + x628);
  Val x1175 = (arg1.bits[113]._super._super * Val(2));
  Val x1176 = (arg1.bits[114]._super._super + x631);
  Val x1177 = (arg1.bits[114]._super._super * Val(2));
  Val x1178 = (arg1.bits[115]._super._super + x634);
  Val x1179 = (arg1.bits[115]._super._super * Val(2));
  Val x1180 = (arg1.bits[116]._super._super + x637);
  Val x1181 = (arg1.bits[116]._super._super * Val(2));
  Val x1182 = (arg1.bits[117]._super._super + x640);
  Val x1183 = (arg1.bits[117]._super._super * Val(2));
  Val x1184 = (arg1.bits[118]._super._super + x643);
  Val x1185 = (arg1.bits[118]._super._super * Val(2));
  Val x1186 = (arg1.bits[119]._super._super + x646);
  Val x1187 = (arg1.bits[119]._super._super * Val(2));
  Val x1188 = (arg1.bits[120]._super._super + x649);
  Val x1189 = (arg1.bits[120]._super._super * Val(2));
  Val x1190 = (arg1.bits[121]._super._super + x652);
  Val x1191 = (arg1.bits[121]._super._super * Val(2));
  Val x1192 = (arg1.bits[122]._super._super + x655);
  Val x1193 = (arg1.bits[122]._super._super * Val(2));
  Val x1194 = (arg1.bits[123]._super._super + x658);
  Val x1195 = (arg1.bits[123]._super._super * Val(2));
  Val x1196 = (arg1.bits[124]._super._super + x661);
  Val x1197 = (arg1.bits[124]._super._super * Val(2));
  Val x1198 = (arg1.bits[125]._super._super + x664);
  Val x1199 = (arg1.bits[125]._super._super * Val(2));
  Val x1200 = (arg1.bits[126]._super._super + x667);
  Val x1201 = (arg1.bits[126]._super._super * Val(2));
  Val x1202 = (arg1.bits[127]._super._super + x670);
  Val x1203 = (arg1.bits[127]._super._super * Val(2));
  Val x1204 = (arg2.bits[96]._super._super + x673);
  Val x1205 = (arg2.bits[96]._super._super * Val(2));
  Val x1206 = (arg2.bits[97]._super._super + x676);
  Val x1207 = (arg2.bits[97]._super._super * Val(2));
  Val x1208 = (arg2.bits[98]._super._super + x679);
  Val x1209 = (arg2.bits[98]._super._super * Val(2));
  Val x1210 = (arg2.bits[99]._super._super + x682);
  Val x1211 = (arg2.bits[99]._super._super * Val(2));
  Val x1212 = (arg1.bits[133]._super._super + x784);
  Val x1213 = (arg1.bits[133]._super._super * Val(2));
  Val x1214 = (arg1.bits[134]._super._super + x787);
  Val x1215 = (arg1.bits[134]._super._super * Val(2));
  Val x1216 = (arg1.bits[135]._super._super + x790);
  Val x1217 = (arg1.bits[135]._super._super * Val(2));
  Val x1218 = (arg1.bits[136]._super._super + x793);
  Val x1219 = (arg1.bits[136]._super._super * Val(2));
  Val x1220 = (arg1.bits[137]._super._super + x796);
  Val x1221 = (arg1.bits[137]._super._super * Val(2));
  Val x1222 = (arg1.bits[138]._super._super + x799);
  Val x1223 = (arg1.bits[138]._super._super * Val(2));
  Val x1224 = (arg1.bits[139]._super._super + x802);
  Val x1225 = (arg1.bits[139]._super._super * Val(2));
  Val x1226 = (arg1.bits[140]._super._super + x805);
  Val x1227 = (arg1.bits[140]._super._super * Val(2));
  Val x1228 = (arg1.bits[141]._super._super + x808);
  Val x1229 = (arg1.bits[141]._super._super * Val(2));
  Val x1230 = (arg1.bits[142]._super._super + x811);
  Val x1231 = (arg1.bits[142]._super._super * Val(2));
  Val x1232 = (arg1.bits[143]._super._super + x814);
  Val x1233 = (arg1.bits[143]._super._super * Val(2));
  Val x1234 = (arg1.bits[144]._super._super + x817);
  Val x1235 = (arg1.bits[144]._super._super * Val(2));
  Val x1236 = (arg1.bits[145]._super._super + x820);
  Val x1237 = (arg1.bits[145]._super._super * Val(2));
  Val x1238 = (arg1.bits[146]._super._super + x823);
  Val x1239 = (arg1.bits[146]._super._super * Val(2));
  Val x1240 = (arg1.bits[147]._super._super + x826);
  Val x1241 = (arg1.bits[147]._super._super * Val(2));
  Val x1242 = (arg1.bits[148]._super._super + x829);
  Val x1243 = (arg1.bits[148]._super._super * Val(2));
  Val x1244 = (arg1.bits[149]._super._super + x832);
  Val x1245 = (arg1.bits[149]._super._super * Val(2));
  Val x1246 = (arg1.bits[150]._super._super + x835);
  Val x1247 = (arg1.bits[150]._super._super * Val(2));
  Val x1248 = (arg1.bits[151]._super._super + x838);
  Val x1249 = (arg1.bits[151]._super._super * Val(2));
  Val x1250 = (arg1.bits[152]._super._super + x841);
  Val x1251 = (arg1.bits[152]._super._super * Val(2));
  Val x1252 = (arg1.bits[153]._super._super + x844);
  Val x1253 = (arg1.bits[153]._super._super * Val(2));
  Val x1254 = (arg1.bits[154]._super._super + x847);
  Val x1255 = (arg1.bits[154]._super._super * Val(2));
  Val x1256 = (arg1.bits[155]._super._super + x850);
  Val x1257 = (arg1.bits[155]._super._super * Val(2));
  Val x1258 = (arg1.bits[156]._super._super + x853);
  Val x1259 = (arg1.bits[156]._super._super * Val(2));
  Val x1260 = (arg1.bits[157]._super._super + x856);
  Val x1261 = (arg1.bits[157]._super._super * Val(2));
  Val x1262 = (arg1.bits[158]._super._super + x859);
  Val x1263 = (arg1.bits[158]._super._super * Val(2));
  Val x1264 = (arg1.bits[159]._super._super + x862);
  Val x1265 = (arg1.bits[159]._super._super * Val(2));
  Val x1266 = (arg2.bits[128]._super._super + x865);
  Val x1267 = (arg2.bits[128]._super._super * Val(2));
  Val x1268 = (arg2.bits[129]._super._super + x868);
  Val x1269 = (arg2.bits[129]._super._super * Val(2));
  Val x1270 = (arg2.bits[130]._super._super + x871);
  Val x1271 = (arg2.bits[130]._super._super * Val(2));
  Val x1272 = (arg2.bits[131]._super._super + x874);
  Val x1273 = (arg2.bits[131]._super._super * Val(2));
  Val x1274 = (arg2.bits[132]._super._super + x877);
  Val x1275 = (arg2.bits[132]._super._super * Val(2));
  Val x1276 = (arg1.bits[160]._super._super + x7);
  Val x1277 = (arg1.bits[160]._super._super * Val(2));
  Val x1278 = (arg1.bits[161]._super._super + x10);
  Val x1279 = (arg1.bits[161]._super._super * Val(2));
  Val x1280 = (arg1.bits[162]._super._super + x13);
  Val x1281 = (arg1.bits[162]._super._super * Val(2));
  Val x1282 = (arg1.bits[163]._super._super + x16);
  Val x1283 = (arg1.bits[163]._super._super * Val(2));
  Val x1284 = (arg1.bits[164]._super._super + x19);
  Val x1285 = (arg1.bits[164]._super._super * Val(2));
  Val x1286 = (arg1.bits[165]._super._super + x22);
  Val x1287 = (arg1.bits[165]._super._super * Val(2));
  Val x1288 = (arg1.bits[166]._super._super + x25);
  Val x1289 = (arg1.bits[166]._super._super * Val(2));
  Val x1290 = (arg1.bits[167]._super._super + x28);
  Val x1291 = (arg1.bits[167]._super._super * Val(2));
  Val x1292 = (arg1.bits[168]._super._super + x31);
  Val x1293 = (arg1.bits[168]._super._super * Val(2));
  Val x1294 = (arg1.bits[169]._super._super + x34);
  Val x1295 = (arg1.bits[169]._super._super * Val(2));
  Val x1296 = (arg1.bits[170]._super._super + x37);
  Val x1297 = (arg1.bits[170]._super._super * Val(2));
  Val x1298 = (arg1.bits[171]._super._super + x40);
  Val x1299 = (arg1.bits[171]._super._super * Val(2));
  Val x1300 = (arg1.bits[172]._super._super + x43);
  Val x1301 = (arg1.bits[172]._super._super * Val(2));
  Val x1302 = (arg1.bits[173]._super._super + x46);
  Val x1303 = (arg1.bits[173]._super._super * Val(2));
  Val x1304 = (arg1.bits[174]._super._super + x49);
  Val x1305 = (arg1.bits[174]._super._super * Val(2));
  Val x1306 = (arg1.bits[175]._super._super + x52);
  Val x1307 = (arg1.bits[175]._super._super * Val(2));
  Val x1308 = (arg1.bits[176]._super._super + x55);
  Val x1309 = (arg1.bits[176]._super._super * Val(2));
  Val x1310 = (arg1.bits[177]._super._super + x58);
  Val x1311 = (arg1.bits[177]._super._super * Val(2));
  Val x1312 = (arg1.bits[178]._super._super + x61);
  Val x1313 = (arg1.bits[178]._super._super * Val(2));
  Val x1314 = (arg1.bits[179]._super._super + x64);
  Val x1315 = (arg1.bits[179]._super._super * Val(2));
  Val x1316 = (arg1.bits[180]._super._super + x67);
  Val x1317 = (arg1.bits[180]._super._super * Val(2));
  Val x1318 = (arg1.bits[181]._super._super + x70);
  Val x1319 = (arg1.bits[181]._super._super * Val(2));
  Val x1320 = (arg1.bits[182]._super._super + x73);
  Val x1321 = (arg1.bits[182]._super._super * Val(2));
  Val x1322 = (arg1.bits[183]._super._super + x76);
  Val x1323 = (arg1.bits[183]._super._super * Val(2));
  Val x1324 = (arg1.bits[184]._super._super + x79);
  Val x1325 = (arg1.bits[184]._super._super * Val(2));
  Val x1326 = (arg1.bits[185]._super._super + x82);
  Val x1327 = (arg1.bits[185]._super._super * Val(2));
  Val x1328 = (arg1.bits[186]._super._super + x85);
  Val x1329 = (arg1.bits[186]._super._super * Val(2));
  Val x1330 = (arg1.bits[187]._super._super + x88);
  Val x1331 = (arg1.bits[187]._super._super * Val(2));
  Val x1332 = (arg2.bits[188]._super._super + x187);
  Val x1333 = (arg2.bits[188]._super._super * Val(2));
  Val x1334 = (arg2.bits[189]._super._super + x190);
  Val x1335 = (arg2.bits[189]._super._super * Val(2));
  Val x1336 = (arg2.bits[190]._super._super + x193);
  Val x1337 = (arg2.bits[190]._super._super * Val(2));
  Val x1338 = (arg2.bits[191]._super._super + x196);
  Val x1339 = (arg2.bits[191]._super._super * Val(2));
  Val x1340 = (arg1.bits[192]._super._super + x199);
  Val x1341 = (arg1.bits[192]._super._super * Val(2));
  Val x1342 = (arg1.bits[193]._super._super + x202);
  Val x1343 = (arg1.bits[193]._super._super * Val(2));
  Val x1344 = (arg1.bits[194]._super._super + x205);
  Val x1345 = (arg1.bits[194]._super._super * Val(2));
  Val x1346 = (arg1.bits[195]._super._super + x208);
  Val x1347 = (arg1.bits[195]._super._super * Val(2));
  Val x1348 = (arg1.bits[196]._super._super + x211);
  Val x1349 = (arg1.bits[196]._super._super * Val(2));
  Val x1350 = (arg1.bits[197]._super._super + x214);
  Val x1351 = (arg1.bits[197]._super._super * Val(2));
  Val x1352 = (arg1.bits[198]._super._super + x217);
  Val x1353 = (arg1.bits[198]._super._super * Val(2));
  Val x1354 = (arg1.bits[199]._super._super + x220);
  Val x1355 = (arg1.bits[199]._super._super * Val(2));
  Val x1356 = (arg1.bits[200]._super._super + x223);
  Val x1357 = (arg1.bits[200]._super._super * Val(2));
  Val x1358 = (arg1.bits[201]._super._super + x226);
  Val x1359 = (arg1.bits[201]._super._super * Val(2));
  Val x1360 = (arg1.bits[202]._super._super + x229);
  Val x1361 = (arg1.bits[202]._super._super * Val(2));
  Val x1362 = (arg1.bits[203]._super._super + x232);
  Val x1363 = (arg1.bits[203]._super._super * Val(2));
  Val x1364 = (arg1.bits[204]._super._super + x235);
  Val x1365 = (arg1.bits[204]._super._super * Val(2));
  Val x1366 = (arg1.bits[205]._super._super + x238);
  Val x1367 = (arg1.bits[205]._super._super * Val(2));
  Val x1368 = (arg1.bits[206]._super._super + x241);
  Val x1369 = (arg1.bits[206]._super._super * Val(2));
  Val x1370 = (arg1.bits[207]._super._super + x244);
  Val x1371 = (arg1.bits[207]._super._super * Val(2));
  Val x1372 = (arg1.bits[208]._super._super + x247);
  Val x1373 = (arg1.bits[208]._super._super * Val(2));
  Val x1374 = (arg1.bits[209]._super._super + x250);
  Val x1375 = (arg1.bits[209]._super._super * Val(2));
  Val x1376 = (arg1.bits[210]._super._super + x253);
  Val x1377 = (arg1.bits[210]._super._super * Val(2));
  Val x1378 = (arg1.bits[211]._super._super + x256);
  Val x1379 = (arg1.bits[211]._super._super * Val(2));
  Val x1380 = (arg2.bits[212]._super._super + x349);
  Val x1381 = (arg2.bits[212]._super._super * Val(2));
  Val x1382 = (arg2.bits[213]._super._super + x352);
  Val x1383 = (arg2.bits[213]._super._super * Val(2));
  Val x1384 = (arg2.bits[214]._super._super + x355);
  Val x1385 = (arg2.bits[214]._super._super * Val(2));
  Val x1386 = (arg2.bits[215]._super._super + x358);
  Val x1387 = (arg2.bits[215]._super._super * Val(2));
  Val x1388 = (arg2.bits[216]._super._super + x361);
  Val x1389 = (arg2.bits[216]._super._super * Val(2));
  Val x1390 = (arg2.bits[217]._super._super + x364);
  Val x1391 = (arg2.bits[217]._super._super * Val(2));
  Val x1392 = (arg2.bits[218]._super._super + x367);
  Val x1393 = (arg2.bits[218]._super._super * Val(2));
  Val x1394 = (arg2.bits[219]._super._super + x370);
  Val x1395 = (arg2.bits[219]._super._super * Val(2));
  Val x1396 = (arg2.bits[220]._super._super + x373);
  Val x1397 = (arg2.bits[220]._super._super * Val(2));
  Val x1398 = (arg2.bits[221]._super._super + x376);
  Val x1399 = (arg2.bits[221]._super._super * Val(2));
  Val x1400 = (arg2.bits[222]._super._super + x379);
  Val x1401 = (arg2.bits[222]._super._super * Val(2));
  Val x1402 = (arg2.bits[223]._super._super + x382);
  Val x1403 = (arg2.bits[223]._super._super * Val(2));
  Val x1404 = (arg1.bits[250]._super._super + x463);
  Val x1405 = (arg1.bits[250]._super._super * Val(2));
  Val x1406 = (arg1.bits[251]._super._super + x466);
  Val x1407 = (arg1.bits[251]._super._super * Val(2));
  Val x1408 = (arg1.bits[252]._super._super + x469);
  Val x1409 = (arg1.bits[252]._super._super * Val(2));
  Val x1410 = (arg1.bits[253]._super._super + x472);
  Val x1411 = (arg1.bits[253]._super._super * Val(2));
  Val x1412 = (arg1.bits[254]._super._super + x475);
  Val x1413 = (arg1.bits[254]._super._super * Val(2));
  Val x1414 = (arg1.bits[255]._super._super + x478);
  Val x1415 = (arg1.bits[255]._super._super * Val(2));
  Val x1416 = (arg2.bits[224]._super._super + x481);
  Val x1417 = (arg2.bits[224]._super._super * Val(2));
  Val x1418 = (arg2.bits[225]._super._super + x484);
  Val x1419 = (arg2.bits[225]._super._super * Val(2));
  Val x1420 = (arg2.bits[226]._super._super + x487);
  Val x1421 = (arg2.bits[226]._super._super * Val(2));
  Val x1422 = (arg2.bits[227]._super._super + x490);
  Val x1423 = (arg2.bits[227]._super._super * Val(2));
  Val x1424 = (arg2.bits[228]._super._super + x493);
  Val x1425 = (arg2.bits[228]._super._super * Val(2));
  Val x1426 = (arg2.bits[229]._super._super + x496);
  Val x1427 = (arg2.bits[229]._super._super * Val(2));
  Val x1428 = (arg2.bits[230]._super._super + x499);
  Val x1429 = (arg2.bits[230]._super._super * Val(2));
  Val x1430 = (arg2.bits[231]._super._super + x502);
  Val x1431 = (arg2.bits[231]._super._super * Val(2));
  Val x1432 = (arg2.bits[232]._super._super + x505);
  Val x1433 = (arg2.bits[232]._super._super * Val(2));
  Val x1434 = (arg2.bits[233]._super._super + x508);
  Val x1435 = (arg2.bits[233]._super._super * Val(2));
  Val x1436 = (arg2.bits[234]._super._super + x511);
  Val x1437 = (arg2.bits[234]._super._super * Val(2));
  Val x1438 = (arg2.bits[235]._super._super + x514);
  Val x1439 = (arg2.bits[235]._super._super * Val(2));
  Val x1440 = (arg2.bits[236]._super._super + x517);
  Val x1441 = (arg2.bits[236]._super._super * Val(2));
  Val x1442 = (arg2.bits[237]._super._super + x520);
  Val x1443 = (arg2.bits[237]._super._super * Val(2));
  Val x1444 = (arg2.bits[238]._super._super + x523);
  Val x1445 = (arg2.bits[238]._super._super * Val(2));
  Val x1446 = (arg2.bits[239]._super._super + x526);
  Val x1447 = (arg2.bits[239]._super._super * Val(2));
  Val x1448 = (arg2.bits[240]._super._super + x529);
  Val x1449 = (arg2.bits[240]._super._super * Val(2));
  Val x1450 = (arg2.bits[241]._super._super + x532);
  Val x1451 = (arg2.bits[241]._super._super * Val(2));
  Val x1452 = (arg2.bits[242]._super._super + x535);
  Val x1453 = (arg2.bits[242]._super._super * Val(2));
  Val x1454 = (arg2.bits[243]._super._super + x538);
  Val x1455 = (arg2.bits[243]._super._super * Val(2));
  Val x1456 = (arg2.bits[244]._super._super + x541);
  Val x1457 = (arg2.bits[244]._super._super * Val(2));
  Val x1458 = (arg2.bits[245]._super._super + x544);
  Val x1459 = (arg2.bits[245]._super._super * Val(2));
  Val x1460 = (arg2.bits[246]._super._super + x547);
  Val x1461 = (arg2.bits[246]._super._super * Val(2));
  Val x1462 = (arg2.bits[247]._super._super + x550);
  Val x1463 = (arg2.bits[247]._super._super * Val(2));
  Val x1464 = (arg2.bits[248]._super._super + x553);
  Val x1465 = (arg2.bits[248]._super._super * Val(2));
  Val x1466 = (arg2.bits[249]._super._super + x556);
  Val x1467 = (arg2.bits[249]._super._super * Val(2));
  Val x1468 = (arg1.bits[256]._super._super + x577);
  Val x1469 = (arg1.bits[256]._super._super * Val(2));
  Val x1470 = (arg1.bits[257]._super._super + x580);
  Val x1471 = (arg1.bits[257]._super._super * Val(2));
  Val x1472 = (arg1.bits[258]._super._super + x583);
  Val x1473 = (arg1.bits[258]._super._super * Val(2));
  Val x1474 = (arg1.bits[259]._super._super + x586);
  Val x1475 = (arg1.bits[259]._super._super * Val(2));
  Val x1476 = (arg1.bits[260]._super._super + x589);
  Val x1477 = (arg1.bits[260]._super._super * Val(2));
  Val x1478 = (arg1.bits[261]._super._super + x592);
  Val x1479 = (arg1.bits[261]._super._super * Val(2));
  Val x1480 = (arg1.bits[262]._super._super + x595);
  Val x1481 = (arg1.bits[262]._super._super * Val(2));
  Val x1482 = (arg1.bits[263]._super._super + x598);
  Val x1483 = (arg1.bits[263]._super._super * Val(2));
  Val x1484 = (arg1.bits[264]._super._super + x601);
  Val x1485 = (arg1.bits[264]._super._super * Val(2));
  Val x1486 = (arg2.bits[265]._super._super + x700);
  Val x1487 = (arg2.bits[265]._super._super * Val(2));
  Val x1488 = (arg2.bits[266]._super._super + x703);
  Val x1489 = (arg2.bits[266]._super._super * Val(2));
  Val x1490 = (arg2.bits[267]._super._super + x706);
  Val x1491 = (arg2.bits[267]._super._super * Val(2));
  Val x1492 = (arg2.bits[268]._super._super + x709);
  Val x1493 = (arg2.bits[268]._super._super * Val(2));
  Val x1494 = (arg2.bits[269]._super._super + x712);
  Val x1495 = (arg2.bits[269]._super._super * Val(2));
  Val x1496 = (arg2.bits[270]._super._super + x715);
  Val x1497 = (arg2.bits[270]._super._super * Val(2));
  Val x1498 = (arg2.bits[271]._super._super + x718);
  Val x1499 = (arg2.bits[271]._super._super * Val(2));
  Val x1500 = (arg2.bits[272]._super._super + x721);
  Val x1501 = (arg2.bits[272]._super._super * Val(2));
  Val x1502 = (arg2.bits[273]._super._super + x724);
  Val x1503 = (arg2.bits[273]._super._super * Val(2));
  Val x1504 = (arg2.bits[274]._super._super + x727);
  Val x1505 = (arg2.bits[274]._super._super * Val(2));
  Val x1506 = (arg2.bits[275]._super._super + x730);
  Val x1507 = (arg2.bits[275]._super._super * Val(2));
  Val x1508 = (arg2.bits[276]._super._super + x733);
  Val x1509 = (arg2.bits[276]._super._super * Val(2));
  Val x1510 = (arg2.bits[277]._super._super + x736);
  Val x1511 = (arg2.bits[277]._super._super * Val(2));
  Val x1512 = (arg2.bits[278]._super._super + x739);
  Val x1513 = (arg2.bits[278]._super._super * Val(2));
  Val x1514 = (arg2.bits[279]._super._super + x742);
  Val x1515 = (arg2.bits[279]._super._super * Val(2));
  Val x1516 = (arg2.bits[280]._super._super + x745);
  Val x1517 = (arg2.bits[280]._super._super * Val(2));
  Val x1518 = (arg2.bits[281]._super._super + x748);
  Val x1519 = (arg2.bits[281]._super._super * Val(2));
  Val x1520 = (arg2.bits[282]._super._super + x751);
  Val x1521 = (arg2.bits[282]._super._super * Val(2));
  Val x1522 = (arg2.bits[283]._super._super + x754);
  Val x1523 = (arg2.bits[283]._super._super * Val(2));
  Val x1524 = (arg2.bits[284]._super._super + x757);
  Val x1525 = (arg2.bits[284]._super._super * Val(2));
  Val x1526 = (arg2.bits[285]._super._super + x760);
  Val x1527 = (arg2.bits[285]._super._super * Val(2));
  Val x1528 = (arg2.bits[286]._super._super + x763);
  Val x1529 = (arg2.bits[286]._super._super * Val(2));
  Val x1530 = (arg2.bits[287]._super._super + x766);
  Val x1531 = (arg2.bits[287]._super._super * Val(2));
  Val x1532 = (arg1.bits[300]._super._super + x805);
  Val x1533 = (arg1.bits[300]._super._super * Val(2));
  Val x1534 = (arg1.bits[301]._super._super + x808);
  Val x1535 = (arg1.bits[301]._super._super * Val(2));
  Val x1536 = (arg1.bits[302]._super._super + x811);
  Val x1537 = (arg1.bits[302]._super._super * Val(2));
  Val x1538 = (arg1.bits[303]._super._super + x814);
  Val x1539 = (arg1.bits[303]._super._super * Val(2));
  Val x1540 = (arg1.bits[304]._super._super + x817);
  Val x1541 = (arg1.bits[304]._super._super * Val(2));
  Val x1542 = (arg1.bits[305]._super._super + x820);
  Val x1543 = (arg1.bits[305]._super._super * Val(2));
  Val x1544 = (arg1.bits[306]._super._super + x823);
  Val x1545 = (arg1.bits[306]._super._super * Val(2));
  Val x1546 = (arg1.bits[307]._super._super + x826);
  Val x1547 = (arg1.bits[307]._super._super * Val(2));
  Val x1548 = (arg1.bits[308]._super._super + x829);
  Val x1549 = (arg1.bits[308]._super._super * Val(2));
  Val x1550 = (arg1.bits[309]._super._super + x832);
  Val x1551 = (arg1.bits[309]._super._super * Val(2));
  Val x1552 = (arg1.bits[310]._super._super + x835);
  Val x1553 = (arg1.bits[310]._super._super * Val(2));
  Val x1554 = (arg1.bits[311]._super._super + x838);
  Val x1555 = (arg1.bits[311]._super._super * Val(2));
  Val x1556 = (arg1.bits[312]._super._super + x841);
  Val x1557 = (arg1.bits[312]._super._super * Val(2));
  Val x1558 = (arg1.bits[313]._super._super + x844);
  Val x1559 = (arg1.bits[313]._super._super * Val(2));
  Val x1560 = (arg1.bits[314]._super._super + x847);
  Val x1561 = (arg1.bits[314]._super._super * Val(2));
  Val x1562 = (arg1.bits[315]._super._super + x850);
  Val x1563 = (arg1.bits[315]._super._super * Val(2));
  Val x1564 = (arg1.bits[316]._super._super + x853);
  Val x1565 = (arg1.bits[316]._super._super * Val(2));
  Val x1566 = (arg1.bits[317]._super._super + x856);
  Val x1567 = (arg1.bits[317]._super._super * Val(2));
  Val x1568 = (arg1.bits[318]._super._super + x859);
  Val x1569 = (arg1.bits[318]._super._super * Val(2));
  Val x1570 = (arg1.bits[319]._super._super + x862);
  Val x1571 = (arg1.bits[319]._super._super * Val(2));
  Val x1572 = (arg2.bits[288]._super._super + x865);
  Val x1573 = (arg2.bits[288]._super._super * Val(2));
  Val x1574 = (arg2.bits[289]._super._super + x868);
  Val x1575 = (arg2.bits[289]._super._super * Val(2));
  Val x1576 = (arg2.bits[290]._super._super + x871);
  Val x1577 = (arg2.bits[290]._super._super * Val(2));
  Val x1578 = (arg2.bits[291]._super._super + x874);
  Val x1579 = (arg2.bits[291]._super._super * Val(2));
  Val x1580 = (arg2.bits[292]._super._super + x877);
  Val x1581 = (arg2.bits[292]._super._super * Val(2));
  Val x1582 = (arg2.bits[293]._super._super + x880);
  Val x1583 = (arg2.bits[293]._super._super * Val(2));
  Val x1584 = (arg2.bits[294]._super._super + x883);
  Val x1585 = (arg2.bits[294]._super._super * Val(2));
  Val x1586 = (arg2.bits[295]._super._super + x886);
  Val x1587 = (arg2.bits[295]._super._super * Val(2));
  Val x1588 = (arg2.bits[296]._super._super + x889);
  Val x1589 = (arg2.bits[296]._super._super * Val(2));
  Val x1590 = (arg2.bits[297]._super._super + x892);
  Val x1591 = (arg2.bits[297]._super._super * Val(2));
  Val x1592 = (arg2.bits[298]._super._super + x895);
  Val x1593 = (arg2.bits[298]._super._super * Val(2));
  Val x1594 = (arg2.bits[299]._super._super + x898);
  Val x1595 = (arg2.bits[299]._super._super * Val(2));
  Val x1596 = (arg1.bits[349]._super._super + x94);
  Val x1597 = (arg1.bits[349]._super._super * Val(2));
  Val x1598 = (arg1.bits[350]._super._super + x97);
  Val x1599 = (arg1.bits[350]._super._super * Val(2));
  Val x1600 = (arg1.bits[351]._super._super + x100);
  Val x1601 = (arg1.bits[351]._super._super * Val(2));
  Val x1602 = (arg2.bits[320]._super._super + x103);
  Val x1603 = (arg2.bits[320]._super._super * Val(2));
  Val x1604 = (arg2.bits[321]._super._super + x106);
  Val x1605 = (arg2.bits[321]._super._super * Val(2));
  Val x1606 = (arg2.bits[322]._super._super + x109);
  Val x1607 = (arg2.bits[322]._super._super * Val(2));
  Val x1608 = (arg2.bits[323]._super._super + x112);
  Val x1609 = (arg2.bits[323]._super._super * Val(2));
  Val x1610 = (arg2.bits[324]._super._super + x115);
  Val x1611 = (arg2.bits[324]._super._super * Val(2));
  Val x1612 = (arg2.bits[325]._super._super + x118);
  Val x1613 = (arg2.bits[325]._super._super * Val(2));
  Val x1614 = (arg2.bits[326]._super._super + x121);
  Val x1615 = (arg2.bits[326]._super._super * Val(2));
  Val x1616 = (arg2.bits[327]._super._super + x124);
  Val x1617 = (arg2.bits[327]._super._super * Val(2));
  Val x1618 = (arg2.bits[328]._super._super + x127);
  Val x1619 = (arg2.bits[328]._super._super * Val(2));
  Val x1620 = (arg2.bits[329]._super._super + x130);
  Val x1621 = (arg2.bits[329]._super._super * Val(2));
  Val x1622 = (arg2.bits[330]._super._super + x133);
  Val x1623 = (arg2.bits[330]._super._super * Val(2));
  Val x1624 = (arg2.bits[331]._super._super + x136);
  Val x1625 = (arg2.bits[331]._super._super * Val(2));
  Val x1626 = (arg2.bits[332]._super._super + x139);
  Val x1627 = (arg2.bits[332]._super._super * Val(2));
  Val x1628 = (arg2.bits[333]._super._super + x142);
  Val x1629 = (arg2.bits[333]._super._super * Val(2));
  Val x1630 = (arg2.bits[334]._super._super + x145);
  Val x1631 = (arg2.bits[334]._super._super * Val(2));
  Val x1632 = (arg2.bits[335]._super._super + x148);
  Val x1633 = (arg2.bits[335]._super._super * Val(2));
  Val x1634 = (arg2.bits[336]._super._super + x151);
  Val x1635 = (arg2.bits[336]._super._super * Val(2));
  Val x1636 = (arg2.bits[337]._super._super + x154);
  Val x1637 = (arg2.bits[337]._super._super * Val(2));
  Val x1638 = (arg2.bits[338]._super._super + x157);
  Val x1639 = (arg2.bits[338]._super._super * Val(2));
  Val x1640 = (arg2.bits[339]._super._super + x160);
  Val x1641 = (arg2.bits[339]._super._super * Val(2));
  Val x1642 = (arg2.bits[340]._super._super + x163);
  Val x1643 = (arg2.bits[340]._super._super * Val(2));
  Val x1644 = (arg2.bits[341]._super._super + x166);
  Val x1645 = (arg2.bits[341]._super._super * Val(2));
  Val x1646 = (arg2.bits[342]._super._super + x169);
  Val x1647 = (arg2.bits[342]._super._super * Val(2));
  Val x1648 = (arg2.bits[343]._super._super + x172);
  Val x1649 = (arg2.bits[343]._super._super * Val(2));
  Val x1650 = (arg2.bits[344]._super._super + x175);
  Val x1651 = (arg2.bits[344]._super._super * Val(2));
  Val x1652 = (arg2.bits[345]._super._super + x178);
  Val x1653 = (arg2.bits[345]._super._super * Val(2));
  Val x1654 = (arg2.bits[346]._super._super + x181);
  Val x1655 = (arg2.bits[346]._super._super * Val(2));
  Val x1656 = (arg2.bits[347]._super._super + x184);
  Val x1657 = (arg2.bits[347]._super._super * Val(2));
  Val x1658 = (arg2.bits[348]._super._super + x187);
  Val x1659 = (arg2.bits[348]._super._super * Val(2));
  Val x1660 = (arg1.bits[374]._super._super + x259);
  Val x1661 = (arg1.bits[374]._super._super * Val(2));
  Val x1662 = (arg1.bits[375]._super._super + x262);
  Val x1663 = (arg1.bits[375]._super._super * Val(2));
  Val x1664 = (arg1.bits[376]._super._super + x265);
  Val x1665 = (arg1.bits[376]._super._super * Val(2));
  Val x1666 = (arg1.bits[377]._super._super + x268);
  Val x1667 = (arg1.bits[377]._super._super * Val(2));
  Val x1668 = (arg1.bits[378]._super._super + x271);
  Val x1669 = (arg1.bits[378]._super._super * Val(2));
  Val x1670 = (arg1.bits[379]._super._super + x274);
  Val x1671 = (arg1.bits[379]._super._super * Val(2));
  Val x1672 = (arg1.bits[380]._super._super + x277);
  Val x1673 = (arg1.bits[380]._super._super * Val(2));
  Val x1674 = (arg1.bits[381]._super._super + x280);
  Val x1675 = (arg1.bits[381]._super._super * Val(2));
  Val x1676 = (arg1.bits[382]._super._super + x283);
  Val x1677 = (arg1.bits[382]._super._super * Val(2));
  Val x1678 = (arg1.bits[383]._super._super + x286);
  Val x1679 = (arg1.bits[383]._super._super * Val(2));
  Val x1680 = (arg2.bits[352]._super._super + x289);
  Val x1681 = (arg2.bits[352]._super._super * Val(2));
  Val x1682 = (arg2.bits[353]._super._super + x292);
  Val x1683 = (arg2.bits[353]._super._super * Val(2));
  Val x1684 = (arg2.bits[354]._super._super + x295);
  Val x1685 = (arg2.bits[354]._super._super * Val(2));
  Val x1686 = (arg2.bits[355]._super._super + x298);
  Val x1687 = (arg2.bits[355]._super._super * Val(2));
  Val x1688 = (arg2.bits[356]._super._super + x301);
  Val x1689 = (arg2.bits[356]._super._super * Val(2));
  Val x1690 = (arg2.bits[357]._super._super + x304);
  Val x1691 = (arg2.bits[357]._super._super * Val(2));
  Val x1692 = (arg2.bits[358]._super._super + x307);
  Val x1693 = (arg2.bits[358]._super._super * Val(2));
  Val x1694 = (arg2.bits[359]._super._super + x310);
  Val x1695 = (arg2.bits[359]._super._super * Val(2));
  Val x1696 = (arg2.bits[360]._super._super + x313);
  Val x1697 = (arg2.bits[360]._super._super * Val(2));
  Val x1698 = (arg2.bits[361]._super._super + x316);
  Val x1699 = (arg2.bits[361]._super._super * Val(2));
  Val x1700 = (arg2.bits[362]._super._super + x319);
  Val x1701 = (arg2.bits[362]._super._super * Val(2));
  Val x1702 = (arg2.bits[363]._super._super + x322);
  Val x1703 = (arg2.bits[363]._super._super * Val(2));
  Val x1704 = (arg2.bits[364]._super._super + x325);
  Val x1705 = (arg2.bits[364]._super._super * Val(2));
  Val x1706 = (arg2.bits[365]._super._super + x328);
  Val x1707 = (arg2.bits[365]._super._super * Val(2));
  Val x1708 = (arg2.bits[366]._super._super + x331);
  Val x1709 = (arg2.bits[366]._super._super * Val(2));
  Val x1710 = (arg2.bits[367]._super._super + x334);
  Val x1711 = (arg2.bits[367]._super._super * Val(2));
  Val x1712 = (arg2.bits[368]._super._super + x337);
  Val x1713 = (arg2.bits[368]._super._super * Val(2));
  Val x1714 = (arg2.bits[369]._super._super + x340);
  Val x1715 = (arg2.bits[369]._super._super * Val(2));
  Val x1716 = (arg2.bits[370]._super._super + x343);
  Val x1717 = (arg2.bits[370]._super._super * Val(2));
  Val x1718 = (arg2.bits[371]._super._super + x346);
  Val x1719 = (arg2.bits[371]._super._super * Val(2));
  Val x1720 = (arg2.bits[372]._super._super + x349);
  Val x1721 = (arg2.bits[372]._super._super * Val(2));
  Val x1722 = (arg2.bits[373]._super._super + x352);
  Val x1723 = (arg2.bits[373]._super._super * Val(2));
  Val x1724 = (arg1.bits[384]._super._super + x385);
  Val x1725 = (arg1.bits[384]._super._super * Val(2));
  Val x1726 = (arg1.bits[385]._super._super + x388);
  Val x1727 = (arg1.bits[385]._super._super * Val(2));
  Val x1728 = (arg1.bits[386]._super._super + x391);
  Val x1729 = (arg1.bits[386]._super._super * Val(2));
  Val x1730 = (arg1.bits[387]._super._super + x394);
  Val x1731 = (arg1.bits[387]._super._super * Val(2));
  Val x1732 = (arg1.bits[388]._super._super + x397);
  Val x1733 = (arg1.bits[388]._super._super * Val(2));
  Val x1734 = (arg1.bits[389]._super._super + x400);
  Val x1735 = (arg1.bits[389]._super._super * Val(2));
  Val x1736 = (arg1.bits[390]._super._super + x403);
  Val x1737 = (arg1.bits[390]._super._super * Val(2));
  Val x1738 = (arg1.bits[391]._super._super + x406);
  Val x1739 = (arg1.bits[391]._super._super * Val(2));
  Val x1740 = (arg1.bits[392]._super._super + x409);
  Val x1741 = (arg1.bits[392]._super._super * Val(2));
  Val x1742 = (arg1.bits[393]._super._super + x412);
  Val x1743 = (arg1.bits[393]._super._super * Val(2));
  Val x1744 = (arg1.bits[394]._super._super + x415);
  Val x1745 = (arg1.bits[394]._super._super * Val(2));
  Val x1746 = (arg1.bits[395]._super._super + x418);
  Val x1747 = (arg1.bits[395]._super._super * Val(2));
  Val x1748 = (arg1.bits[396]._super._super + x421);
  Val x1749 = (arg1.bits[396]._super._super * Val(2));
  Val x1750 = (arg1.bits[397]._super._super + x424);
  Val x1751 = (arg1.bits[397]._super._super * Val(2));
  Val x1752 = (arg1.bits[398]._super._super + x427);
  Val x1753 = (arg1.bits[398]._super._super * Val(2));
  Val x1754 = (arg1.bits[399]._super._super + x430);
  Val x1755 = (arg1.bits[399]._super._super * Val(2));
  Val x1756 = (arg1.bits[400]._super._super + x433);
  Val x1757 = (arg1.bits[400]._super._super * Val(2));
  Val x1758 = (arg1.bits[401]._super._super + x436);
  Val x1759 = (arg1.bits[401]._super._super * Val(2));
  Val x1760 = (arg1.bits[402]._super._super + x439);
  Val x1761 = (arg1.bits[402]._super._super * Val(2));
  Val x1762 = (arg1.bits[403]._super._super + x442);
  Val x1763 = (arg1.bits[403]._super._super * Val(2));
  Val x1764 = (arg1.bits[404]._super._super + x445);
  Val x1765 = (arg1.bits[404]._super._super * Val(2));
  Val x1766 = (arg2.bits[405]._super._super + x544);
  Val x1767 = (arg2.bits[405]._super._super * Val(2));
  Val x1768 = (arg2.bits[406]._super._super + x547);
  Val x1769 = (arg2.bits[406]._super._super * Val(2));
  Val x1770 = (arg2.bits[407]._super._super + x550);
  Val x1771 = (arg2.bits[407]._super._super * Val(2));
  Val x1772 = (arg2.bits[408]._super._super + x553);
  Val x1773 = (arg2.bits[408]._super._super * Val(2));
  Val x1774 = (arg2.bits[409]._super._super + x556);
  Val x1775 = (arg2.bits[409]._super._super * Val(2));
  Val x1776 = (arg2.bits[410]._super._super + x559);
  Val x1777 = (arg2.bits[410]._super._super * Val(2));
  Val x1778 = (arg2.bits[411]._super._super + x562);
  Val x1779 = (arg2.bits[411]._super._super * Val(2));
  Val x1780 = (arg2.bits[412]._super._super + x565);
  Val x1781 = (arg2.bits[412]._super._super * Val(2));
  Val x1782 = (arg2.bits[413]._super._super + x568);
  Val x1783 = (arg2.bits[413]._super._super * Val(2));
  Val x1784 = (arg2.bits[414]._super._super + x571);
  Val x1785 = (arg2.bits[414]._super._super * Val(2));
  Val x1786 = (arg2.bits[415]._super._super + x574);
  Val x1787 = (arg2.bits[415]._super._super * Val(2));
  Val x1788 = (arg1.bits[423]._super._super + x598);
  Val x1789 = (arg1.bits[423]._super._super * Val(2));
  Val x1790 = (arg1.bits[424]._super._super + x601);
  Val x1791 = (arg1.bits[424]._super._super * Val(2));
  Val x1792 = (arg1.bits[425]._super._super + x604);
  Val x1793 = (arg1.bits[425]._super._super * Val(2));
  Val x1794 = (arg1.bits[426]._super._super + x607);
  Val x1795 = (arg1.bits[426]._super._super * Val(2));
  Val x1796 = (arg1.bits[427]._super._super + x610);
  Val x1797 = (arg1.bits[427]._super._super * Val(2));
  Val x1798 = (arg1.bits[428]._super._super + x613);
  Val x1799 = (arg1.bits[428]._super._super * Val(2));
  Val x1800 = (arg1.bits[429]._super._super + x616);
  Val x1801 = (arg1.bits[429]._super._super * Val(2));
  Val x1802 = (arg1.bits[430]._super._super + x619);
  Val x1803 = (arg1.bits[430]._super._super * Val(2));
  Val x1804 = (arg1.bits[431]._super._super + x622);
  Val x1805 = (arg1.bits[431]._super._super * Val(2));
  Val x1806 = (arg1.bits[432]._super._super + x625);
  Val x1807 = (arg1.bits[432]._super._super * Val(2));
  Val x1808 = (arg1.bits[433]._super._super + x628);
  Val x1809 = (arg1.bits[433]._super._super * Val(2));
  Val x1810 = (arg1.bits[434]._super._super + x631);
  Val x1811 = (arg1.bits[434]._super._super * Val(2));
  Val x1812 = (arg1.bits[435]._super._super + x634);
  Val x1813 = (arg1.bits[435]._super._super * Val(2));
  Val x1814 = (arg1.bits[436]._super._super + x637);
  Val x1815 = (arg1.bits[436]._super._super * Val(2));
  Val x1816 = (arg1.bits[437]._super._super + x640);
  Val x1817 = (arg1.bits[437]._super._super * Val(2));
  Val x1818 = (arg1.bits[438]._super._super + x643);
  Val x1819 = (arg1.bits[438]._super._super * Val(2));
  Val x1820 = (arg1.bits[439]._super._super + x646);
  Val x1821 = (arg1.bits[439]._super._super * Val(2));
  Val x1822 = (arg1.bits[440]._super._super + x649);
  Val x1823 = (arg1.bits[440]._super._super * Val(2));
  Val x1824 = (arg1.bits[441]._super._super + x652);
  Val x1825 = (arg1.bits[441]._super._super * Val(2));
  Val x1826 = (arg1.bits[442]._super._super + x655);
  Val x1827 = (arg1.bits[442]._super._super * Val(2));
  Val x1828 = (arg1.bits[443]._super._super + x658);
  Val x1829 = (arg1.bits[443]._super._super * Val(2));
  Val x1830 = (arg1.bits[444]._super._super + x661);
  Val x1831 = (arg1.bits[444]._super._super * Val(2));
  Val x1832 = (arg1.bits[445]._super._super + x664);
  Val x1833 = (arg1.bits[445]._super._super * Val(2));
  Val x1834 = (arg1.bits[446]._super._super + x667);
  Val x1835 = (arg1.bits[446]._super._super * Val(2));
  Val x1836 = (arg1.bits[447]._super._super + x670);
  Val x1837 = (arg1.bits[447]._super._super * Val(2));
  Val x1838 = (arg2.bits[416]._super._super + x673);
  Val x1839 = (arg2.bits[416]._super._super * Val(2));
  Val x1840 = (arg2.bits[417]._super._super + x676);
  Val x1841 = (arg2.bits[417]._super._super * Val(2));
  Val x1842 = (arg2.bits[418]._super._super + x679);
  Val x1843 = (arg2.bits[418]._super._super * Val(2));
  Val x1844 = (arg2.bits[419]._super._super + x682);
  Val x1845 = (arg2.bits[419]._super._super * Val(2));
  Val x1846 = (arg2.bits[420]._super._super + x685);
  Val x1847 = (arg2.bits[420]._super._super * Val(2));
  Val x1848 = (arg2.bits[421]._super._super + x688);
  Val x1849 = (arg2.bits[421]._super._super * Val(2));
  Val x1850 = (arg2.bits[422]._super._super + x691);
  Val x1851 = (arg2.bits[422]._super._super * Val(2));
  Val x1852 = (arg1.bits[448]._super._super + x769);
  Val x1853 = (arg1.bits[448]._super._super * Val(2));
  Val x1854 = (arg1.bits[449]._super._super + x772);
  Val x1855 = (arg1.bits[449]._super._super * Val(2));
  Val x1856 = (arg1.bits[450]._super._super + x775);
  Val x1857 = (arg1.bits[450]._super._super * Val(2));
  Val x1858 = (arg1.bits[451]._super._super + x778);
  Val x1859 = (arg1.bits[451]._super._super * Val(2));
  Val x1860 = (arg1.bits[452]._super._super + x781);
  Val x1861 = (arg1.bits[452]._super._super * Val(2));
  Val x1862 = (arg1.bits[453]._super._super + x784);
  Val x1863 = (arg1.bits[453]._super._super * Val(2));
  Val x1864 = (arg1.bits[454]._super._super + x787);
  Val x1865 = (arg1.bits[454]._super._super * Val(2));
  Val x1866 = (arg1.bits[455]._super._super + x790);
  Val x1867 = (arg1.bits[455]._super._super * Val(2));
  Val x1868 = (arg1.bits[456]._super._super + x793);
  Val x1869 = (arg1.bits[456]._super._super * Val(2));
  Val x1870 = (arg1.bits[457]._super._super + x796);
  Val x1871 = (arg1.bits[457]._super._super * Val(2));
  Val x1872 = (arg1.bits[458]._super._super + x799);
  Val x1873 = (arg1.bits[458]._super._super * Val(2));
  Val x1874 = (arg1.bits[459]._super._super + x802);
  Val x1875 = (arg1.bits[459]._super._super * Val(2));
  Val x1876 = (arg1.bits[460]._super._super + x805);
  Val x1877 = (arg1.bits[460]._super._super * Val(2));
  Val x1878 = (arg1.bits[461]._super._super + x808);
  Val x1879 = (arg1.bits[461]._super._super * Val(2));
  Val x1880 = (arg1.bits[462]._super._super + x811);
  Val x1881 = (arg1.bits[462]._super._super * Val(2));
  Val x1882 = (arg1.bits[463]._super._super + x814);
  Val x1883 = (arg1.bits[463]._super._super * Val(2));
  Val x1884 = (arg1.bits[464]._super._super + x817);
  Val x1885 = (arg1.bits[464]._super._super * Val(2));
  Val x1886 = (arg1.bits[465]._super._super + x820);
  Val x1887 = (arg1.bits[465]._super._super * Val(2));
  Val x1888 = (arg1.bits[466]._super._super + x823);
  Val x1889 = (arg1.bits[466]._super._super * Val(2));
  Val x1890 = (arg1.bits[467]._super._super + x826);
  Val x1891 = (arg1.bits[467]._super._super * Val(2));
  Val x1892 = (arg1.bits[468]._super._super + x829);
  Val x1893 = (arg1.bits[468]._super._super * Val(2));
  Val x1894 = (arg1.bits[469]._super._super + x832);
  Val x1895 = (arg1.bits[469]._super._super * Val(2));
  Val x1896 = (arg1.bits[470]._super._super + x835);
  Val x1897 = (arg1.bits[470]._super._super * Val(2));
  Val x1898 = (arg1.bits[471]._super._super + x838);
  Val x1899 = (arg1.bits[471]._super._super * Val(2));
  Val x1900 = (arg1.bits[472]._super._super + x841);
  Val x1901 = (arg1.bits[472]._super._super * Val(2));
  Val x1902 = (arg2.bits[473]._super._super + x937);
  Val x1903 = (arg2.bits[473]._super._super * Val(2));
  Val x1904 = (arg2.bits[474]._super._super + x940);
  Val x1905 = (arg2.bits[474]._super._super * Val(2));
  Val x1906 = (arg2.bits[475]._super._super + x943);
  Val x1907 = (arg2.bits[475]._super._super * Val(2));
  Val x1908 = (arg2.bits[476]._super._super + x946);
  Val x1909 = (arg2.bits[476]._super._super * Val(2));
  Val x1910 = (arg2.bits[477]._super._super + x949);
  Val x1911 = (arg2.bits[477]._super._super * Val(2));
  Val x1912 = (arg2.bits[478]._super._super + x952);
  Val x1913 = (arg2.bits[478]._super._super * Val(2));
  Val x1914 = (arg2.bits[479]._super._super + x955);
  Val x1915 = (arg2.bits[479]._super._super * Val(2));
  Val x1916 = (arg1.bits[480]._super._super + x7);
  Val x1917 = (arg1.bits[480]._super._super * Val(2));
  Val x1918 = (arg1.bits[481]._super._super + x10);
  Val x1919 = (arg1.bits[481]._super._super * Val(2));
  Val x1920 = (arg1.bits[482]._super._super + x13);
  Val x1921 = (arg1.bits[482]._super._super * Val(2));
  Val x1922 = (arg1.bits[483]._super._super + x16);
  Val x1923 = (arg1.bits[483]._super._super * Val(2));
  Val x1924 = (arg1.bits[484]._super._super + x19);
  Val x1925 = (arg1.bits[484]._super._super * Val(2));
  Val x1926 = (arg1.bits[485]._super._super + x22);
  Val x1927 = (arg1.bits[485]._super._super * Val(2));
  Val x1928 = (arg1.bits[486]._super._super + x25);
  Val x1929 = (arg1.bits[486]._super._super * Val(2));
  Val x1930 = (arg1.bits[487]._super._super + x28);
  Val x1931 = (arg1.bits[487]._super._super * Val(2));
  Val x1932 = (arg1.bits[488]._super._super + x31);
  Val x1933 = (arg1.bits[488]._super._super * Val(2));
  Val x1934 = (arg1.bits[489]._super._super + x34);
  Val x1935 = (arg1.bits[489]._super._super * Val(2));
  Val x1936 = (arg1.bits[490]._super._super + x37);
  Val x1937 = (arg1.bits[490]._super._super * Val(2));
  Val x1938 = (arg1.bits[491]._super._super + x40);
  Val x1939 = (arg1.bits[491]._super._super * Val(2));
  Val x1940 = (arg1.bits[492]._super._super + x43);
  Val x1941 = (arg1.bits[492]._super._super * Val(2));
  Val x1942 = (arg1.bits[493]._super._super + x46);
  Val x1943 = (arg1.bits[493]._super._super * Val(2));
  Val x1944 = (arg1.bits[494]._super._super + x49);
  Val x1945 = (arg1.bits[494]._super._super * Val(2));
  Val x1946 = (arg1.bits[495]._super._super + x52);
  Val x1947 = (arg1.bits[495]._super._super * Val(2));
  Val x1948 = (arg1.bits[496]._super._super + x55);
  Val x1949 = (arg1.bits[496]._super._super * Val(2));
  Val x1950 = (arg1.bits[497]._super._super + x58);
  Val x1951 = (arg1.bits[497]._super._super * Val(2));
  Val x1952 = (arg1.bits[498]._super._super + x61);
  Val x1953 = (arg1.bits[498]._super._super * Val(2));
  Val x1954 = (arg1.bits[499]._super._super + x64);
  Val x1955 = (arg1.bits[499]._super._super * Val(2));
  Val x1956 = (arg1.bits[500]._super._super + x67);
  Val x1957 = (arg1.bits[500]._super._super * Val(2));
  Val x1958 = (arg1.bits[501]._super._super + x70);
  Val x1959 = (arg1.bits[501]._super._super * Val(2));
  Val x1960 = (arg1.bits[502]._super._super + x73);
  Val x1961 = (arg1.bits[502]._super._super * Val(2));
  Val x1962 = (arg2.bits[503]._super._super + x172);
  Val x1963 = (arg2.bits[503]._super._super * Val(2));
  Val x1964 = (arg2.bits[504]._super._super + x175);
  Val x1965 = (arg2.bits[504]._super._super * Val(2));
  Val x1966 = (arg2.bits[505]._super._super + x178);
  Val x1967 = (arg2.bits[505]._super._super * Val(2));
  Val x1968 = (arg2.bits[506]._super._super + x181);
  Val x1969 = (arg2.bits[506]._super._super * Val(2));
  Val x1970 = (arg2.bits[507]._super._super + x184);
  Val x1971 = (arg2.bits[507]._super._super * Val(2));
  Val x1972 = (arg2.bits[508]._super._super + x187);
  Val x1973 = (arg2.bits[508]._super._super * Val(2));
  Val x1974 = (arg2.bits[509]._super._super + x190);
  Val x1975 = (arg2.bits[509]._super._super * Val(2));
  Val x1976 = (arg2.bits[510]._super._super + x193);
  Val x1977 = (arg2.bits[510]._super._super * Val(2));
  Val x1978 = (arg2.bits[511]._super._super + x196);
  Val x1979 = (arg2.bits[511]._super._super * Val(2));
  Val x1980 = (arg1.bits[512]._super._super + x199);
  Val x1981 = (arg1.bits[512]._super._super * Val(2));
  Val x1982 = (arg1.bits[513]._super._super + x202);
  Val x1983 = (arg1.bits[513]._super._super * Val(2));
  Val x1984 = (arg1.bits[514]._super._super + x205);
  Val x1985 = (arg1.bits[514]._super._super * Val(2));
  Val x1986 = (arg1.bits[515]._super._super + x208);
  Val x1987 = (arg1.bits[515]._super._super * Val(2));
  Val x1988 = (arg1.bits[516]._super._super + x211);
  Val x1989 = (arg1.bits[516]._super._super * Val(2));
  Val x1990 = (arg1.bits[517]._super._super + x214);
  Val x1991 = (arg1.bits[517]._super._super * Val(2));
  Val x1992 = (arg1.bits[518]._super._super + x217);
  Val x1993 = (arg1.bits[518]._super._super * Val(2));
  Val x1994 = (arg1.bits[519]._super._super + x220);
  Val x1995 = (arg1.bits[519]._super._super * Val(2));
  Val x1996 = (arg1.bits[520]._super._super + x223);
  Val x1997 = (arg1.bits[520]._super._super * Val(2));
  Val x1998 = (arg1.bits[521]._super._super + x226);
  Val x1999 = (arg1.bits[521]._super._super * Val(2));
  Val x2000 = (arg1.bits[522]._super._super + x229);
  Val x2001 = (arg1.bits[522]._super._super * Val(2));
  Val x2002 = (arg1.bits[523]._super._super + x232);
  Val x2003 = (arg1.bits[523]._super._super * Val(2));
  Val x2004 = (arg1.bits[524]._super._super + x235);
  Val x2005 = (arg1.bits[524]._super._super * Val(2));
  Val x2006 = (arg1.bits[525]._super._super + x238);
  Val x2007 = (arg1.bits[525]._super._super * Val(2));
  Val x2008 = (arg1.bits[526]._super._super + x241);
  Val x2009 = (arg1.bits[526]._super._super * Val(2));
  Val x2010 = (arg1.bits[527]._super._super + x244);
  Val x2011 = (arg1.bits[527]._super._super * Val(2));
  Val x2012 = (arg1.bits[528]._super._super + x247);
  Val x2013 = (arg1.bits[528]._super._super * Val(2));
  Val x2014 = (arg1.bits[529]._super._super + x250);
  Val x2015 = (arg1.bits[529]._super._super * Val(2));
  Val x2016 = (arg1.bits[530]._super._super + x253);
  Val x2017 = (arg1.bits[530]._super._super * Val(2));
  Val x2018 = (arg2.bits[531]._super._super + x346);
  Val x2019 = (arg2.bits[531]._super._super * Val(2));
  Val x2020 = (arg2.bits[532]._super._super + x349);
  Val x2021 = (arg2.bits[532]._super._super * Val(2));
  Val x2022 = (arg2.bits[533]._super._super + x352);
  Val x2023 = (arg2.bits[533]._super._super * Val(2));
  Val x2024 = (arg2.bits[534]._super._super + x355);
  Val x2025 = (arg2.bits[534]._super._super * Val(2));
  Val x2026 = (arg2.bits[535]._super._super + x358);
  Val x2027 = (arg2.bits[535]._super._super * Val(2));
  Val x2028 = (arg2.bits[536]._super._super + x361);
  Val x2029 = (arg2.bits[536]._super._super * Val(2));
  Val x2030 = (arg2.bits[537]._super._super + x364);
  Val x2031 = (arg2.bits[537]._super._super * Val(2));
  Val x2032 = (arg2.bits[538]._super._super + x367);
  Val x2033 = (arg2.bits[538]._super._super * Val(2));
  Val x2034 = (arg2.bits[539]._super._super + x370);
  Val x2035 = (arg2.bits[539]._super._super * Val(2));
  Val x2036 = (arg2.bits[540]._super._super + x373);
  Val x2037 = (arg2.bits[540]._super._super * Val(2));
  Val x2038 = (arg2.bits[541]._super._super + x376);
  Val x2039 = (arg2.bits[541]._super._super * Val(2));
  Val x2040 = (arg2.bits[542]._super._super + x379);
  Val x2041 = (arg2.bits[542]._super._super * Val(2));
  Val x2042 = (arg2.bits[543]._super._super + x382);
  Val x2043 = (arg2.bits[543]._super._super * Val(2));
  Val x2044 = (arg1.bits[561]._super._super + x436);
  Val x2045 = (arg1.bits[561]._super._super * Val(2));
  Val x2046 = (arg1.bits[562]._super._super + x439);
  Val x2047 = (arg1.bits[562]._super._super * Val(2));
  Val x2048 = (arg1.bits[563]._super._super + x442);
  Val x2049 = (arg1.bits[563]._super._super * Val(2));
  Val x2050 = (arg1.bits[564]._super._super + x445);
  Val x2051 = (arg1.bits[564]._super._super * Val(2));
  Val x2052 = (arg1.bits[565]._super._super + x448);
  Val x2053 = (arg1.bits[565]._super._super * Val(2));
  Val x2054 = (arg1.bits[566]._super._super + x451);
  Val x2055 = (arg1.bits[566]._super._super * Val(2));
  Val x2056 = (arg1.bits[567]._super._super + x454);
  Val x2057 = (arg1.bits[567]._super._super * Val(2));
  Val x2058 = (arg1.bits[568]._super._super + x457);
  Val x2059 = (arg1.bits[568]._super._super * Val(2));
  Val x2060 = (arg1.bits[569]._super._super + x460);
  Val x2061 = (arg1.bits[569]._super._super * Val(2));
  Val x2062 = (arg1.bits[570]._super._super + x463);
  Val x2063 = (arg1.bits[570]._super._super * Val(2));
  Val x2064 = (arg1.bits[571]._super._super + x466);
  Val x2065 = (arg1.bits[571]._super._super * Val(2));
  Val x2066 = (arg1.bits[572]._super._super + x469);
  Val x2067 = (arg1.bits[572]._super._super * Val(2));
  Val x2068 = (arg1.bits[573]._super._super + x472);
  Val x2069 = (arg1.bits[573]._super._super * Val(2));
  Val x2070 = (arg1.bits[574]._super._super + x475);
  Val x2071 = (arg1.bits[574]._super._super * Val(2));
  Val x2072 = (arg1.bits[575]._super._super + x478);
  Val x2073 = (arg1.bits[575]._super._super * Val(2));
  Val x2074 = (arg2.bits[544]._super._super + x481);
  Val x2075 = (arg2.bits[544]._super._super * Val(2));
  Val x2076 = (arg2.bits[545]._super._super + x484);
  Val x2077 = (arg2.bits[545]._super._super * Val(2));
  Val x2078 = (arg2.bits[546]._super._super + x487);
  Val x2079 = (arg2.bits[546]._super._super * Val(2));
  Val x2080 = (arg2.bits[547]._super._super + x490);
  Val x2081 = (arg2.bits[547]._super._super * Val(2));
  Val x2082 = (arg2.bits[548]._super._super + x493);
  Val x2083 = (arg2.bits[548]._super._super * Val(2));
  Val x2084 = (arg2.bits[549]._super._super + x496);
  Val x2085 = (arg2.bits[549]._super._super * Val(2));
  Val x2086 = (arg2.bits[550]._super._super + x499);
  Val x2087 = (arg2.bits[550]._super._super * Val(2));
  Val x2088 = (arg2.bits[551]._super._super + x502);
  Val x2089 = (arg2.bits[551]._super._super * Val(2));
  Val x2090 = (arg2.bits[552]._super._super + x505);
  Val x2091 = (arg2.bits[552]._super._super * Val(2));
  Val x2092 = (arg2.bits[553]._super._super + x508);
  Val x2093 = (arg2.bits[553]._super._super * Val(2));
  Val x2094 = (arg2.bits[554]._super._super + x511);
  Val x2095 = (arg2.bits[554]._super._super * Val(2));
  Val x2096 = (arg2.bits[555]._super._super + x514);
  Val x2097 = (arg2.bits[555]._super._super * Val(2));
  Val x2098 = (arg2.bits[556]._super._super + x517);
  Val x2099 = (arg2.bits[556]._super._super * Val(2));
  Val x2100 = (arg2.bits[557]._super._super + x520);
  Val x2101 = (arg2.bits[557]._super._super * Val(2));
  Val x2102 = (arg2.bits[558]._super._super + x523);
  Val x2103 = (arg2.bits[558]._super._super * Val(2));
  Val x2104 = (arg2.bits[559]._super._super + x526);
  Val x2105 = (arg2.bits[559]._super._super * Val(2));
  Val x2106 = (arg2.bits[560]._super._super + x529);
  Val x2107 = (arg2.bits[560]._super._super * Val(2));
  Val x2108 = (arg1.bits[587]._super._super + x610);
  Val x2109 = (arg1.bits[587]._super._super * Val(2));
  Val x2110 = (arg1.bits[588]._super._super + x613);
  Val x2111 = (arg1.bits[588]._super._super * Val(2));
  Val x2112 = (arg1.bits[589]._super._super + x616);
  Val x2113 = (arg1.bits[589]._super._super * Val(2));
  Val x2114 = (arg1.bits[590]._super._super + x619);
  Val x2115 = (arg1.bits[590]._super._super * Val(2));
  Val x2116 = (arg1.bits[591]._super._super + x622);
  Val x2117 = (arg1.bits[591]._super._super * Val(2));
  Val x2118 = (arg1.bits[592]._super._super + x625);
  Val x2119 = (arg1.bits[592]._super._super * Val(2));
  Val x2120 = (arg1.bits[593]._super._super + x628);
  Val x2121 = (arg1.bits[593]._super._super * Val(2));
  Val x2122 = (arg1.bits[594]._super._super + x631);
  Val x2123 = (arg1.bits[594]._super._super * Val(2));
  Val x2124 = (arg1.bits[595]._super._super + x634);
  Val x2125 = (arg1.bits[595]._super._super * Val(2));
  Val x2126 = (arg1.bits[596]._super._super + x637);
  Val x2127 = (arg1.bits[596]._super._super * Val(2));
  Val x2128 = (arg1.bits[597]._super._super + x640);
  Val x2129 = (arg1.bits[597]._super._super * Val(2));
  Val x2130 = (arg1.bits[598]._super._super + x643);
  Val x2131 = (arg1.bits[598]._super._super * Val(2));
  Val x2132 = (arg1.bits[599]._super._super + x646);
  Val x2133 = (arg1.bits[599]._super._super * Val(2));
  Val x2134 = (arg1.bits[600]._super._super + x649);
  Val x2135 = (arg1.bits[600]._super._super * Val(2));
  Val x2136 = (arg1.bits[601]._super._super + x652);
  Val x2137 = (arg1.bits[601]._super._super * Val(2));
  Val x2138 = (arg1.bits[602]._super._super + x655);
  Val x2139 = (arg1.bits[602]._super._super * Val(2));
  Val x2140 = (arg1.bits[603]._super._super + x658);
  Val x2141 = (arg1.bits[603]._super._super * Val(2));
  Val x2142 = (arg1.bits[604]._super._super + x661);
  Val x2143 = (arg1.bits[604]._super._super * Val(2));
  Val x2144 = (arg1.bits[605]._super._super + x664);
  Val x2145 = (arg1.bits[605]._super._super * Val(2));
  Val x2146 = (arg1.bits[606]._super._super + x667);
  Val x2147 = (arg1.bits[606]._super._super * Val(2));
  Val x2148 = (arg1.bits[607]._super._super + x670);
  Val x2149 = (arg1.bits[607]._super._super * Val(2));
  Val x2150 = (arg2.bits[576]._super._super + x673);
  Val x2151 = (arg2.bits[576]._super._super * Val(2));
  Val x2152 = (arg2.bits[577]._super._super + x676);
  Val x2153 = (arg2.bits[577]._super._super * Val(2));
  Val x2154 = (arg2.bits[578]._super._super + x679);
  Val x2155 = (arg2.bits[578]._super._super * Val(2));
  Val x2156 = (arg2.bits[579]._super._super + x682);
  Val x2157 = (arg2.bits[579]._super._super * Val(2));
  Val x2158 = (arg2.bits[580]._super._super + x685);
  Val x2159 = (arg2.bits[580]._super._super * Val(2));
  Val x2160 = (arg2.bits[581]._super._super + x688);
  Val x2161 = (arg2.bits[581]._super._super * Val(2));
  Val x2162 = (arg2.bits[582]._super._super + x691);
  Val x2163 = (arg2.bits[582]._super._super * Val(2));
  Val x2164 = (arg2.bits[583]._super._super + x694);
  Val x2165 = (arg2.bits[583]._super._super * Val(2));
  Val x2166 = (arg2.bits[584]._super._super + x697);
  Val x2167 = (arg2.bits[584]._super._super * Val(2));
  Val x2168 = (arg2.bits[585]._super._super + x700);
  Val x2169 = (arg2.bits[585]._super._super * Val(2));
  Val x2170 = (arg2.bits[586]._super._super + x703);
  Val x2171 = (arg2.bits[586]._super._super * Val(2));
  Val x2172 = (arg1.bits[632]._super._super + x841);
  Val x2173 = (arg1.bits[632]._super._super * Val(2));
  Val x2174 = (arg1.bits[633]._super._super + x844);
  Val x2175 = (arg1.bits[633]._super._super * Val(2));
  Val x2176 = (arg1.bits[634]._super._super + x847);
  Val x2177 = (arg1.bits[634]._super._super * Val(2));
  Val x2178 = (arg1.bits[635]._super._super + x850);
  Val x2179 = (arg1.bits[635]._super._super * Val(2));
  Val x2180 = (arg1.bits[636]._super._super + x853);
  Val x2181 = (arg1.bits[636]._super._super * Val(2));
  Val x2182 = (arg1.bits[637]._super._super + x856);
  Val x2183 = (arg1.bits[637]._super._super * Val(2));
  Val x2184 = (arg1.bits[638]._super._super + x859);
  Val x2185 = (arg1.bits[638]._super._super * Val(2));
  Val x2186 = (arg1.bits[639]._super._super + x862);
  Val x2187 = (arg1.bits[639]._super._super * Val(2));
  Val x2188 = (arg2.bits[608]._super._super + x865);
  Val x2189 = (arg2.bits[608]._super._super * Val(2));
  Val x2190 = (arg2.bits[609]._super._super + x868);
  Val x2191 = (arg2.bits[609]._super._super * Val(2));
  Val x2192 = (arg2.bits[610]._super._super + x871);
  Val x2193 = (arg2.bits[610]._super._super * Val(2));
  Val x2194 = (arg2.bits[611]._super._super + x874);
  Val x2195 = (arg2.bits[611]._super._super * Val(2));
  Val x2196 = (arg2.bits[612]._super._super + x877);
  Val x2197 = (arg2.bits[612]._super._super * Val(2));
  Val x2198 = (arg2.bits[613]._super._super + x880);
  Val x2199 = (arg2.bits[613]._super._super * Val(2));
  Val x2200 = (arg2.bits[614]._super._super + x883);
  Val x2201 = (arg2.bits[614]._super._super * Val(2));
  Val x2202 = (arg2.bits[615]._super._super + x886);
  Val x2203 = (arg2.bits[615]._super._super * Val(2));
  Val x2204 = (arg2.bits[616]._super._super + x889);
  Val x2205 = (arg2.bits[616]._super._super * Val(2));
  Val x2206 = (arg2.bits[617]._super._super + x892);
  Val x2207 = (arg2.bits[617]._super._super * Val(2));
  Val x2208 = (arg2.bits[618]._super._super + x895);
  Val x2209 = (arg2.bits[618]._super._super * Val(2));
  Val x2210 = (arg2.bits[619]._super._super + x898);
  Val x2211 = (arg2.bits[619]._super._super * Val(2));
  Val x2212 = (arg2.bits[620]._super._super + x901);
  Val x2213 = (arg2.bits[620]._super._super * Val(2));
  Val x2214 = (arg2.bits[621]._super._super + x904);
  Val x2215 = (arg2.bits[621]._super._super * Val(2));
  Val x2216 = (arg2.bits[622]._super._super + x907);
  Val x2217 = (arg2.bits[622]._super._super * Val(2));
  Val x2218 = (arg2.bits[623]._super._super + x910);
  Val x2219 = (arg2.bits[623]._super._super * Val(2));
  Val x2220 = (arg2.bits[624]._super._super + x913);
  Val x2221 = (arg2.bits[624]._super._super * Val(2));
  Val x2222 = (arg2.bits[625]._super._super + x916);
  Val x2223 = (arg2.bits[625]._super._super * Val(2));
  Val x2224 = (arg2.bits[626]._super._super + x919);
  Val x2225 = (arg2.bits[626]._super._super * Val(2));
  Val x2226 = (arg2.bits[627]._super._super + x922);
  Val x2227 = (arg2.bits[627]._super._super * Val(2));
  Val x2228 = (arg2.bits[628]._super._super + x925);
  Val x2229 = (arg2.bits[628]._super._super * Val(2));
  Val x2230 = (arg2.bits[629]._super._super + x928);
  Val x2231 = (arg2.bits[629]._super._super * Val(2));
  Val x2232 = (arg2.bits[630]._super._super + x931);
  Val x2233 = (arg2.bits[630]._super._super * Val(2));
  Val x2234 = (arg2.bits[631]._super._super + x934);
  Val x2235 = (arg2.bits[631]._super._super * Val(2));
  Val x2236 = (arg1.bits[654]._super._super + x49);
  Val x2237 = (arg1.bits[654]._super._super * Val(2));
  Val x2238 = (arg1.bits[655]._super._super + x52);
  Val x2239 = (arg1.bits[655]._super._super * Val(2));
  Val x2240 = (arg1.bits[656]._super._super + x55);
  Val x2241 = (arg1.bits[656]._super._super * Val(2));
  Val x2242 = (arg1.bits[657]._super._super + x58);
  Val x2243 = (arg1.bits[657]._super._super * Val(2));
  Val x2244 = (arg1.bits[658]._super._super + x61);
  Val x2245 = (arg1.bits[658]._super._super * Val(2));
  Val x2246 = (arg1.bits[659]._super._super + x64);
  Val x2247 = (arg1.bits[659]._super._super * Val(2));
  Val x2248 = (arg1.bits[660]._super._super + x67);
  Val x2249 = (arg1.bits[660]._super._super * Val(2));
  Val x2250 = (arg1.bits[661]._super._super + x70);
  Val x2251 = (arg1.bits[661]._super._super * Val(2));
  Val x2252 = (arg1.bits[662]._super._super + x73);
  Val x2253 = (arg1.bits[662]._super._super * Val(2));
  Val x2254 = (arg1.bits[663]._super._super + x76);
  Val x2255 = (arg1.bits[663]._super._super * Val(2));
  Val x2256 = (arg1.bits[664]._super._super + x79);
  Val x2257 = (arg1.bits[664]._super._super * Val(2));
  Val x2258 = (arg1.bits[665]._super._super + x82);
  Val x2259 = (arg1.bits[665]._super._super * Val(2));
  Val x2260 = (arg1.bits[666]._super._super + x85);
  Val x2261 = (arg1.bits[666]._super._super * Val(2));
  Val x2262 = (arg1.bits[667]._super._super + x88);
  Val x2263 = (arg1.bits[667]._super._super * Val(2));
  Val x2264 = (arg1.bits[668]._super._super + x91);
  Val x2265 = (arg1.bits[668]._super._super * Val(2));
  Val x2266 = (arg1.bits[669]._super._super + x94);
  Val x2267 = (arg1.bits[669]._super._super * Val(2));
  Val x2268 = (arg1.bits[670]._super._super + x97);
  Val x2269 = (arg1.bits[670]._super._super * Val(2));
  Val x2270 = (arg1.bits[671]._super._super + x100);
  Val x2271 = (arg1.bits[671]._super._super * Val(2));
  Val x2272 = (arg2.bits[640]._super._super + x103);
  Val x2273 = (arg2.bits[640]._super._super * Val(2));
  Val x2274 = (arg2.bits[641]._super._super + x106);
  Val x2275 = (arg2.bits[641]._super._super * Val(2));
  Val x2276 = (arg2.bits[642]._super._super + x109);
  Val x2277 = (arg2.bits[642]._super._super * Val(2));
  Val x2278 = (arg2.bits[643]._super._super + x112);
  Val x2279 = (arg2.bits[643]._super._super * Val(2));
  Val x2280 = (arg2.bits[644]._super._super + x115);
  Val x2281 = (arg2.bits[644]._super._super * Val(2));
  Val x2282 = (arg2.bits[645]._super._super + x118);
  Val x2283 = (arg2.bits[645]._super._super * Val(2));
  Val x2284 = (arg2.bits[646]._super._super + x121);
  Val x2285 = (arg2.bits[646]._super._super * Val(2));
  Val x2286 = (arg2.bits[647]._super._super + x124);
  Val x2287 = (arg2.bits[647]._super._super * Val(2));
  Val x2288 = (arg2.bits[648]._super._super + x127);
  Val x2289 = (arg2.bits[648]._super._super * Val(2));
  Val x2290 = (arg2.bits[649]._super._super + x130);
  Val x2291 = (arg2.bits[649]._super._super * Val(2));
  Val x2292 = (arg2.bits[650]._super._super + x133);
  Val x2293 = (arg2.bits[650]._super._super * Val(2));
  Val x2294 = (arg2.bits[651]._super._super + x136);
  Val x2295 = (arg2.bits[651]._super._super * Val(2));
  Val x2296 = (arg2.bits[652]._super._super + x139);
  Val x2297 = (arg2.bits[652]._super._super * Val(2));
  Val x2298 = (arg2.bits[653]._super._super + x142);
  Val x2299 = (arg2.bits[653]._super._super * Val(2));
  Val x2300 = (arg1.bits[702]._super._super + x283);
  Val x2301 = (arg1.bits[702]._super._super * Val(2));
  Val x2302 = (arg1.bits[703]._super._super + x286);
  Val x2303 = (arg1.bits[703]._super._super * Val(2));
  Val x2304 = (arg2.bits[672]._super._super + x289);
  Val x2305 = (arg2.bits[672]._super._super * Val(2));
  Val x2306 = (arg2.bits[673]._super._super + x292);
  Val x2307 = (arg2.bits[673]._super._super * Val(2));
  Val x2308 = (arg2.bits[674]._super._super + x295);
  Val x2309 = (arg2.bits[674]._super._super * Val(2));
  Val x2310 = (arg2.bits[675]._super._super + x298);
  Val x2311 = (arg2.bits[675]._super._super * Val(2));
  Val x2312 = (arg2.bits[676]._super._super + x301);
  Val x2313 = (arg2.bits[676]._super._super * Val(2));
  Val x2314 = (arg2.bits[677]._super._super + x304);
  Val x2315 = (arg2.bits[677]._super._super * Val(2));
  Val x2316 = (arg2.bits[678]._super._super + x307);
  Val x2317 = (arg2.bits[678]._super._super * Val(2));
  Val x2318 = (arg2.bits[679]._super._super + x310);
  Val x2319 = (arg2.bits[679]._super._super * Val(2));
  Val x2320 = (arg2.bits[680]._super._super + x313);
  Val x2321 = (arg2.bits[680]._super._super * Val(2));
  Val x2322 = (arg2.bits[681]._super._super + x316);
  Val x2323 = (arg2.bits[681]._super._super * Val(2));
  Val x2324 = (arg2.bits[682]._super._super + x319);
  Val x2325 = (arg2.bits[682]._super._super * Val(2));
  Val x2326 = (arg2.bits[683]._super._super + x322);
  Val x2327 = (arg2.bits[683]._super._super * Val(2));
  Val x2328 = (arg2.bits[684]._super._super + x325);
  Val x2329 = (arg2.bits[684]._super._super * Val(2));
  Val x2330 = (arg2.bits[685]._super._super + x328);
  Val x2331 = (arg2.bits[685]._super._super * Val(2));
  Val x2332 = (arg2.bits[686]._super._super + x331);
  Val x2333 = (arg2.bits[686]._super._super * Val(2));
  Val x2334 = (arg2.bits[687]._super._super + x334);
  Val x2335 = (arg2.bits[687]._super._super * Val(2));
  Val x2336 = (arg2.bits[688]._super._super + x337);
  Val x2337 = (arg2.bits[688]._super._super * Val(2));
  Val x2338 = (arg2.bits[689]._super._super + x340);
  Val x2339 = (arg2.bits[689]._super._super * Val(2));
  Val x2340 = (arg2.bits[690]._super._super + x343);
  Val x2341 = (arg2.bits[690]._super._super * Val(2));
  Val x2342 = (arg2.bits[691]._super._super + x346);
  Val x2343 = (arg2.bits[691]._super._super * Val(2));
  Val x2344 = (arg2.bits[692]._super._super + x349);
  Val x2345 = (arg2.bits[692]._super._super * Val(2));
  Val x2346 = (arg2.bits[693]._super._super + x352);
  Val x2347 = (arg2.bits[693]._super._super * Val(2));
  Val x2348 = (arg2.bits[694]._super._super + x355);
  Val x2349 = (arg2.bits[694]._super._super * Val(2));
  Val x2350 = (arg2.bits[695]._super._super + x358);
  Val x2351 = (arg2.bits[695]._super._super * Val(2));
  Val x2352 = (arg2.bits[696]._super._super + x361);
  Val x2353 = (arg2.bits[696]._super._super * Val(2));
  Val x2354 = (arg2.bits[697]._super._super + x364);
  Val x2355 = (arg2.bits[697]._super._super * Val(2));
  Val x2356 = (arg2.bits[698]._super._super + x367);
  Val x2357 = (arg2.bits[698]._super._super * Val(2));
  Val x2358 = (arg2.bits[699]._super._super + x370);
  Val x2359 = (arg2.bits[699]._super._super * Val(2));
  Val x2360 = (arg2.bits[700]._super._super + x373);
  Val x2361 = (arg2.bits[700]._super._super * Val(2));
  Val x2362 = (arg2.bits[701]._super._super + x376);
  Val x2363 = (arg2.bits[701]._super._super * Val(2));
  Val x2364 = (arg1.bits[704]._super._super + x385);
  Val x2365 = (arg1.bits[704]._super._super * Val(2));
  Val x2366 = (arg1.bits[705]._super._super + x388);
  Val x2367 = (arg1.bits[705]._super._super * Val(2));
  Val x2368 = (arg1.bits[706]._super._super + x391);
  Val x2369 = (arg1.bits[706]._super._super * Val(2));
  Val x2370 = (arg2.bits[707]._super._super + x490);
  Val x2371 = (arg2.bits[707]._super._super * Val(2));
  Val x2372 = (arg2.bits[708]._super._super + x493);
  Val x2373 = (arg2.bits[708]._super._super * Val(2));
  Val x2374 = (arg2.bits[709]._super._super + x496);
  Val x2375 = (arg2.bits[709]._super._super * Val(2));
  Val x2376 = (arg2.bits[710]._super._super + x499);
  Val x2377 = (arg2.bits[710]._super._super * Val(2));
  Val x2378 = (arg2.bits[711]._super._super + x502);
  Val x2379 = (arg2.bits[711]._super._super * Val(2));
  Val x2380 = (arg2.bits[712]._super._super + x505);
  Val x2381 = (arg2.bits[712]._super._super * Val(2));
  Val x2382 = (arg2.bits[713]._super._super + x508);
  Val x2383 = (arg2.bits[713]._super._super * Val(2));
  Val x2384 = (arg2.bits[714]._super._super + x511);
  Val x2385 = (arg2.bits[714]._super._super * Val(2));
  Val x2386 = (arg2.bits[715]._super._super + x514);
  Val x2387 = (arg2.bits[715]._super._super * Val(2));
  Val x2388 = (arg2.bits[716]._super._super + x517);
  Val x2389 = (arg2.bits[716]._super._super * Val(2));
  Val x2390 = (arg2.bits[717]._super._super + x520);
  Val x2391 = (arg2.bits[717]._super._super * Val(2));
  Val x2392 = (arg2.bits[718]._super._super + x523);
  Val x2393 = (arg2.bits[718]._super._super * Val(2));
  Val x2394 = (arg2.bits[719]._super._super + x526);
  Val x2395 = (arg2.bits[719]._super._super * Val(2));
  Val x2396 = (arg2.bits[720]._super._super + x529);
  Val x2397 = (arg2.bits[720]._super._super * Val(2));
  Val x2398 = (arg2.bits[721]._super._super + x532);
  Val x2399 = (arg2.bits[721]._super._super * Val(2));
  Val x2400 = (arg2.bits[722]._super._super + x535);
  Val x2401 = (arg2.bits[722]._super._super * Val(2));
  Val x2402 = (arg2.bits[723]._super._super + x538);
  Val x2403 = (arg2.bits[723]._super._super * Val(2));
  Val x2404 = (arg2.bits[724]._super._super + x541);
  Val x2405 = (arg2.bits[724]._super._super * Val(2));
  Val x2406 = (arg2.bits[725]._super._super + x544);
  Val x2407 = (arg2.bits[725]._super._super * Val(2));
  Val x2408 = (arg2.bits[726]._super._super + x547);
  Val x2409 = (arg2.bits[726]._super._super * Val(2));
  Val x2410 = (arg2.bits[727]._super._super + x550);
  Val x2411 = (arg2.bits[727]._super._super * Val(2));
  Val x2412 = (arg2.bits[728]._super._super + x553);
  Val x2413 = (arg2.bits[728]._super._super * Val(2));
  Val x2414 = (arg2.bits[729]._super._super + x556);
  Val x2415 = (arg2.bits[729]._super._super * Val(2));
  Val x2416 = (arg2.bits[730]._super._super + x559);
  Val x2417 = (arg2.bits[730]._super._super * Val(2));
  Val x2418 = (arg2.bits[731]._super._super + x562);
  Val x2419 = (arg2.bits[731]._super._super * Val(2));
  Val x2420 = (arg2.bits[732]._super._super + x565);
  Val x2421 = (arg2.bits[732]._super._super * Val(2));
  Val x2422 = (arg2.bits[733]._super._super + x568);
  Val x2423 = (arg2.bits[733]._super._super * Val(2));
  Val x2424 = (arg2.bits[734]._super._super + x571);
  Val x2425 = (arg2.bits[734]._super._super * Val(2));
  Val x2426 = (arg2.bits[735]._super._super + x574);
  Val x2427 = (arg2.bits[735]._super._super * Val(2));
  Val x2428 = (arg1.bits[736]._super._super + x577);
  Val x2429 = (arg1.bits[736]._super._super * Val(2));
  Val x2430 = (arg1.bits[737]._super._super + x580);
  Val x2431 = (arg1.bits[737]._super._super * Val(2));
  Val x2432 = (arg1.bits[738]._super._super + x583);
  Val x2433 = (arg1.bits[738]._super._super * Val(2));
  Val x2434 = (arg1.bits[739]._super._super + x586);
  Val x2435 = (arg1.bits[739]._super._super * Val(2));
  Val x2436 = (arg1.bits[740]._super._super + x589);
  Val x2437 = (arg1.bits[740]._super._super * Val(2));
  Val x2438 = (arg1.bits[741]._super._super + x592);
  Val x2439 = (arg1.bits[741]._super._super * Val(2));
  Val x2440 = (arg1.bits[742]._super._super + x595);
  Val x2441 = (arg1.bits[742]._super._super * Val(2));
  Val x2442 = (arg1.bits[743]._super._super + x598);
  Val x2443 = (arg1.bits[743]._super._super * Val(2));
  Val x2444 = (arg2.bits[744]._super._super + x697);
  Val x2445 = (arg2.bits[744]._super._super * Val(2));
  Val x2446 = (arg2.bits[745]._super._super + x700);
  Val x2447 = (arg2.bits[745]._super._super * Val(2));
  Val x2448 = (arg2.bits[746]._super._super + x703);
  Val x2449 = (arg2.bits[746]._super._super * Val(2));
  Val x2450 = (arg2.bits[747]._super._super + x706);
  Val x2451 = (arg2.bits[747]._super._super * Val(2));
  Val x2452 = (arg2.bits[748]._super._super + x709);
  Val x2453 = (arg2.bits[748]._super._super * Val(2));
  Val x2454 = (arg2.bits[749]._super._super + x712);
  Val x2455 = (arg2.bits[749]._super._super * Val(2));
  Val x2456 = (arg2.bits[750]._super._super + x715);
  Val x2457 = (arg2.bits[750]._super._super * Val(2));
  Val x2458 = (arg2.bits[751]._super._super + x718);
  Val x2459 = (arg2.bits[751]._super._super * Val(2));
  Val x2460 = (arg2.bits[752]._super._super + x721);
  Val x2461 = (arg2.bits[752]._super._super * Val(2));
  Val x2462 = (arg2.bits[753]._super._super + x724);
  Val x2463 = (arg2.bits[753]._super._super * Val(2));
  Val x2464 = (arg2.bits[754]._super._super + x727);
  Val x2465 = (arg2.bits[754]._super._super * Val(2));
  Val x2466 = (arg2.bits[755]._super._super + x730);
  Val x2467 = (arg2.bits[755]._super._super * Val(2));
  Val x2468 = (arg2.bits[756]._super._super + x733);
  Val x2469 = (arg2.bits[756]._super._super * Val(2));
  Val x2470 = (arg2.bits[757]._super._super + x736);
  Val x2471 = (arg2.bits[757]._super._super * Val(2));
  Val x2472 = (arg2.bits[758]._super._super + x739);
  Val x2473 = (arg2.bits[758]._super._super * Val(2));
  Val x2474 = (arg2.bits[759]._super._super + x742);
  Val x2475 = (arg2.bits[759]._super._super * Val(2));
  Val x2476 = (arg2.bits[760]._super._super + x745);
  Val x2477 = (arg2.bits[760]._super._super * Val(2));
  Val x2478 = (arg2.bits[761]._super._super + x748);
  Val x2479 = (arg2.bits[761]._super._super * Val(2));
  Val x2480 = (arg2.bits[762]._super._super + x751);
  Val x2481 = (arg2.bits[762]._super._super * Val(2));
  Val x2482 = (arg2.bits[763]._super._super + x754);
  Val x2483 = (arg2.bits[763]._super._super * Val(2));
  Val x2484 = (arg2.bits[764]._super._super + x757);
  Val x2485 = (arg2.bits[764]._super._super * Val(2));
  Val x2486 = (arg2.bits[765]._super._super + x760);
  Val x2487 = (arg2.bits[765]._super._super * Val(2));
  Val x2488 = (arg2.bits[766]._super._super + x763);
  Val x2489 = (arg2.bits[766]._super._super * Val(2));
  Val x2490 = (arg2.bits[767]._super._super + x766);
  Val x2491 = (arg2.bits[767]._super._super * Val(2));
  Val x2492 = (arg1.bits[786]._super._super + x823);
  Val x2493 = (arg1.bits[786]._super._super * Val(2));
  Val x2494 = (arg1.bits[787]._super._super + x826);
  Val x2495 = (arg1.bits[787]._super._super * Val(2));
  Val x2496 = (arg1.bits[788]._super._super + x829);
  Val x2497 = (arg1.bits[788]._super._super * Val(2));
  Val x2498 = (arg1.bits[789]._super._super + x832);
  Val x2499 = (arg1.bits[789]._super._super * Val(2));
  Val x2500 = (arg1.bits[790]._super._super + x835);
  Val x2501 = (arg1.bits[790]._super._super * Val(2));
  Val x2502 = (arg1.bits[791]._super._super + x838);
  Val x2503 = (arg1.bits[791]._super._super * Val(2));
  Val x2504 = (arg1.bits[792]._super._super + x841);
  Val x2505 = (arg1.bits[792]._super._super * Val(2));
  Val x2506 = (arg1.bits[793]._super._super + x844);
  Val x2507 = (arg1.bits[793]._super._super * Val(2));
  Val x2508 = (arg1.bits[794]._super._super + x847);
  Val x2509 = (arg1.bits[794]._super._super * Val(2));
  Val x2510 = (arg1.bits[795]._super._super + x850);
  Val x2511 = (arg1.bits[795]._super._super * Val(2));
  Val x2512 = (arg1.bits[796]._super._super + x853);
  Val x2513 = (arg1.bits[796]._super._super * Val(2));
  Val x2514 = (arg1.bits[797]._super._super + x856);
  Val x2515 = (arg1.bits[797]._super._super * Val(2));
  Val x2516 = (arg1.bits[798]._super._super + x859);
  Val x2517 = (arg1.bits[798]._super._super * Val(2));
  Val x2518 = (arg1.bits[799]._super._super + x862);
  Val x2519 = (arg1.bits[799]._super._super * Val(2));
  Val x2520 = (arg2.bits[768]._super._super + x865);
  Val x2521 = (arg2.bits[768]._super._super * Val(2));
  Val x2522 = (arg2.bits[769]._super._super + x868);
  Val x2523 = (arg2.bits[769]._super._super * Val(2));
  Val x2524 = (arg2.bits[770]._super._super + x871);
  Val x2525 = (arg2.bits[770]._super._super * Val(2));
  Val x2526 = (arg2.bits[771]._super._super + x874);
  Val x2527 = (arg2.bits[771]._super._super * Val(2));
  Val x2528 = (arg2.bits[772]._super._super + x877);
  Val x2529 = (arg2.bits[772]._super._super * Val(2));
  Val x2530 = (arg2.bits[773]._super._super + x880);
  Val x2531 = (arg2.bits[773]._super._super * Val(2));
  Val x2532 = (arg2.bits[774]._super._super + x883);
  Val x2533 = (arg2.bits[774]._super._super * Val(2));
  Val x2534 = (arg2.bits[775]._super._super + x886);
  Val x2535 = (arg2.bits[775]._super._super * Val(2));
  Val x2536 = (arg2.bits[776]._super._super + x889);
  Val x2537 = (arg2.bits[776]._super._super * Val(2));
  Val x2538 = (arg2.bits[777]._super._super + x892);
  Val x2539 = (arg2.bits[777]._super._super * Val(2));
  Val x2540 = (arg2.bits[778]._super._super + x895);
  Val x2541 = (arg2.bits[778]._super._super * Val(2));
  Val x2542 = (arg2.bits[779]._super._super + x898);
  Val x2543 = (arg2.bits[779]._super._super * Val(2));
  Val x2544 = (arg2.bits[780]._super._super + x901);
  Val x2545 = (arg2.bits[780]._super._super * Val(2));
  Val x2546 = (arg2.bits[781]._super._super + x904);
  Val x2547 = (arg2.bits[781]._super._super * Val(2));
  Val x2548 = (arg2.bits[782]._super._super + x907);
  Val x2549 = (arg2.bits[782]._super._super * Val(2));
  Val x2550 = (arg2.bits[783]._super._super + x910);
  Val x2551 = (arg2.bits[783]._super._super * Val(2));
  Val x2552 = (arg2.bits[784]._super._super + x913);
  Val x2553 = (arg2.bits[784]._super._super * Val(2));
  Val x2554 = (arg2.bits[785]._super._super + x916);
  Val x2555 = (arg2.bits[785]._super._super * Val(2));
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:107)
  Val100Array x2556 = Val100Array{
      arg3.kflat[0]._super._super,  arg3.kflat[1]._super._super,  arg3.kflat[2]._super._super,
      arg3.kflat[3]._super._super,  arg3.kflat[4]._super._super,  arg3.kflat[5]._super._super,
      arg3.kflat[6]._super._super,  arg3.kflat[7]._super._super,  arg3.kflat[8]._super._super,
      arg3.kflat[9]._super._super,  arg3.kflat[10]._super._super, arg3.kflat[11]._super._super,
      arg3.kflat[12]._super._super, arg3.kflat[13]._super._super, arg3.kflat[14]._super._super,
      arg3.kflat[15]._super._super, arg3.kflat[16]._super._super, arg3.kflat[17]._super._super,
      arg3.kflat[18]._super._super, arg3.kflat[19]._super._super, arg3.kflat[20]._super._super,
      arg3.kflat[21]._super._super, arg3.kflat[22]._super._super, arg3.kflat[23]._super._super,
      arg3.kflat[24]._super._super, arg3.kflat[25]._super._super, arg3.kflat[26]._super._super,
      arg3.kflat[27]._super._super, arg3.kflat[28]._super._super, arg3.kflat[29]._super._super,
      arg3.kflat[30]._super._super, arg3.kflat[31]._super._super, arg3.kflat[32]._super._super,
      arg3.kflat[33]._super._super, arg3.kflat[34]._super._super, arg3.kflat[35]._super._super,
      arg3.kflat[36]._super._super, arg3.kflat[37]._super._super, arg3.kflat[38]._super._super,
      arg3.kflat[39]._super._super, arg3.kflat[40]._super._super, arg3.kflat[41]._super._super,
      arg3.kflat[42]._super._super, arg3.kflat[43]._super._super, arg3.kflat[44]._super._super,
      arg3.kflat[45]._super._super, arg3.kflat[46]._super._super, arg3.kflat[47]._super._super,
      arg3.kflat[48]._super._super, arg3.kflat[49]._super._super, arg3.kflat[50]._super._super,
      arg3.kflat[51]._super._super, arg3.kflat[52]._super._super, arg3.kflat[53]._super._super,
      arg3.kflat[54]._super._super, arg3.kflat[55]._super._super, arg3.kflat[56]._super._super,
      arg3.kflat[57]._super._super, arg3.kflat[58]._super._super, arg3.kflat[59]._super._super,
      arg3.kflat[60]._super._super, arg3.kflat[61]._super._super, arg3.kflat[62]._super._super,
      arg3.kflat[63]._super._super, arg3.kflat[64]._super._super, arg3.kflat[65]._super._super,
      arg3.kflat[66]._super._super, arg3.kflat[67]._super._super, arg3.kflat[68]._super._super,
      arg3.kflat[69]._super._super, arg3.kflat[70]._super._super, arg3.kflat[71]._super._super,
      arg3.kflat[72]._super._super, arg3.kflat[73]._super._super, arg3.kflat[74]._super._super,
      arg3.kflat[75]._super._super, arg3.kflat[76]._super._super, arg3.kflat[77]._super._super,
      arg3.kflat[78]._super._super, arg3.kflat[79]._super._super, arg3.kflat[80]._super._super,
      arg3.kflat[81]._super._super, arg3.kflat[82]._super._super, arg3.kflat[83]._super._super,
      arg3.kflat[84]._super._super, arg3.kflat[85]._super._super, arg3.kflat[86]._super._super,
      arg3.kflat[87]._super._super, arg3.kflat[88]._super._super, arg3.kflat[89]._super._super,
      arg3.kflat[90]._super._super, arg3.kflat[91]._super._super, arg3.kflat[92]._super._super,
      arg3.kflat[93]._super._super, arg3.kflat[94]._super._super, arg3.kflat[95]._super._super,
      arg3.kflat[96]._super._super, arg3.kflat[97]._super._super, arg3.kflat[98]._super._super,
      arg3.kflat[99]._super._super};
  Val16Array x2557 = Val16Array{arg3.sflat[0]._super,
                                arg3.sflat[1]._super,
                                arg3.sflat[2]._super,
                                arg3.sflat[3]._super,
                                arg3.sflat[4]._super,
                                arg3.sflat[5]._super,
                                arg3.sflat[6]._super,
                                arg3.sflat[7]._super,
                                arg3.sflat[8]._super,
                                arg3.sflat[9]._super,
                                arg3.sflat[10]._super,
                                arg3.sflat[11]._super,
                                arg3.sflat[12]._super,
                                arg3.sflat[13]._super,
                                arg3.sflat[14]._super,
                                arg3.sflat[15]._super};
  // KeccakRound12(zirgen/circuit/keccak2/top.zir:111)
  TopStateStruct x2558 = exec_TopState(
      ctx,
      Val800Array{(x956 - (x957 * x103)),   (x958 - (x959 * x106)),   (x960 - (x961 * x109)),
                  (x962 - (x963 * x112)),   (x964 - (x965 * x115)),   (x966 - (x967 * x118)),
                  (x968 - (x969 * x121)),   (x970 - (x971 * x124)),   (x972 - (x973 * x127)),
                  (x974 - (x975 * x130)),   (x976 - (x977 * x133)),   (x978 - (x979 * x136)),
                  (x980 - (x981 * x139)),   (x982 - (x983 * x142)),   (x984 - (x985 * x145)),
                  (x986 - (x987 * x148)),   (x988 - (x989 * x151)),   (x990 - (x991 * x154)),
                  (x992 - (x993 * x157)),   (x994 - (x995 * x160)),   (x996 - (x997 * x163)),
                  (x998 - (x999 * x166)),   (x1000 - (x1001 * x169)), (x1002 - (x1003 * x172)),
                  (x1004 - (x1005 * x175)), (x1006 - (x1007 * x178)), (x1008 - (x1009 * x181)),
                  (x1010 - (x1011 * x184)), (x1012 - (x1013 * x187)), (x1014 - (x1015 * x190)),
                  (x1016 - (x1017 * x193)), (x1018 - (x1019 * x196)), (x1380 - (x1381 * x349)),
                  (x1382 - (x1383 * x352)), (x1384 - (x1385 * x355)), (x1386 - (x1387 * x358)),
                  (x1388 - (x1389 * x361)), (x1390 - (x1391 * x364)), (x1392 - (x1393 * x367)),
                  (x1394 - (x1395 * x370)), (x1396 - (x1397 * x373)), (x1398 - (x1399 * x376)),
                  (x1400 - (x1401 * x379)), (x1402 - (x1403 * x382)), (x1340 - (x1341 * x199)),
                  (x1342 - (x1343 * x202)), (x1344 - (x1345 * x205)), (x1346 - (x1347 * x208)),
                  (x1348 - (x1349 * x211)), (x1350 - (x1351 * x214)), (x1352 - (x1353 * x217)),
                  (x1354 - (x1355 * x220)), (x1356 - (x1357 * x223)), (x1358 - (x1359 * x226)),
                  (x1360 - (x1361 * x229)), (x1362 - (x1363 * x232)), (x1364 - (x1365 * x235)),
                  (x1366 - (x1367 * x238)), (x1368 - (x1369 * x241)), (x1370 - (x1371 * x244)),
                  (x1372 - (x1373 * x247)), (x1374 - (x1375 * x250)), (x1376 - (x1377 * x253)),
                  (x1378 - (x1379 * x256)), (x1766 - (x1767 * x544)), (x1768 - (x1769 * x547)),
                  (x1770 - (x1771 * x550)), (x1772 - (x1773 * x553)), (x1774 - (x1775 * x556)),
                  (x1776 - (x1777 * x559)), (x1778 - (x1779 * x562)), (x1780 - (x1781 * x565)),
                  (x1782 - (x1783 * x568)), (x1784 - (x1785 * x571)), (x1786 - (x1787 * x574)),
                  (x1724 - (x1725 * x385)), (x1726 - (x1727 * x388)), (x1728 - (x1729 * x391)),
                  (x1730 - (x1731 * x394)), (x1732 - (x1733 * x397)), (x1734 - (x1735 * x400)),
                  (x1736 - (x1737 * x403)), (x1738 - (x1739 * x406)), (x1740 - (x1741 * x409)),
                  (x1742 - (x1743 * x412)), (x1744 - (x1745 * x415)), (x1746 - (x1747 * x418)),
                  (x1748 - (x1749 * x421)), (x1750 - (x1751 * x424)), (x1752 - (x1753 * x427)),
                  (x1754 - (x1755 * x430)), (x1756 - (x1757 * x433)), (x1758 - (x1759 * x436)),
                  (x1760 - (x1761 * x439)), (x1762 - (x1763 * x442)), (x1764 - (x1765 * x445)),
                  (x2108 - (x2109 * x610)), (x2110 - (x2111 * x613)), (x2112 - (x2113 * x616)),
                  (x2114 - (x2115 * x619)), (x2116 - (x2117 * x622)), (x2118 - (x2119 * x625)),
                  (x2120 - (x2121 * x628)), (x2122 - (x2123 * x631)), (x2124 - (x2125 * x634)),
                  (x2126 - (x2127 * x637)), (x2128 - (x2129 * x640)), (x2130 - (x2131 * x643)),
                  (x2132 - (x2133 * x646)), (x2134 - (x2135 * x649)), (x2136 - (x2137 * x652)),
                  (x2138 - (x2139 * x655)), (x2140 - (x2141 * x658)), (x2142 - (x2143 * x661)),
                  (x2144 - (x2145 * x664)), (x2146 - (x2147 * x667)), (x2148 - (x2149 * x670)),
                  (x2150 - (x2151 * x673)), (x2152 - (x2153 * x676)), (x2154 - (x2155 * x679)),
                  (x2156 - (x2157 * x682)), (x2158 - (x2159 * x685)), (x2160 - (x2161 * x688)),
                  (x2162 - (x2163 * x691)), (x2164 - (x2165 * x694)), (x2166 - (x2167 * x697)),
                  (x2168 - (x2169 * x700)), (x2170 - (x2171 * x703)), (x2492 - (x2493 * x823)),
                  (x2494 - (x2495 * x826)), (x2496 - (x2497 * x829)), (x2498 - (x2499 * x832)),
                  (x2500 - (x2501 * x835)), (x2502 - (x2503 * x838)), (x2504 - (x2505 * x841)),
                  (x2506 - (x2507 * x844)), (x2508 - (x2509 * x847)), (x2510 - (x2511 * x850)),
                  (x2512 - (x2513 * x853)), (x2514 - (x2515 * x856)), (x2516 - (x2517 * x859)),
                  (x2518 - (x2519 * x862)), (x2520 - (x2521 * x865)), (x2522 - (x2523 * x868)),
                  (x2524 - (x2525 * x871)), (x2526 - (x2527 * x874)), (x2528 - (x2529 * x877)),
                  (x2530 - (x2531 * x880)), (x2532 - (x2533 * x883)), (x2534 - (x2535 * x886)),
                  (x2536 - (x2537 * x889)), (x2538 - (x2539 * x892)), (x2540 - (x2541 * x895)),
                  (x2542 - (x2543 * x898)), (x2544 - (x2545 * x901)), (x2546 - (x2547 * x904)),
                  (x2548 - (x2549 * x907)), (x2550 - (x2551 * x910)), (x2552 - (x2553 * x913)),
                  (x2554 - (x2555 * x916)), (x1148 - (x1149 * x589)), (x1150 - (x1151 * x592)),
                  (x1152 - (x1153 * x595)), (x1154 - (x1155 * x598)), (x1156 - (x1157 * x601)),
                  (x1158 - (x1159 * x604)), (x1160 - (x1161 * x607)), (x1162 - (x1163 * x610)),
                  (x1164 - (x1165 * x613)), (x1166 - (x1167 * x616)), (x1168 - (x1169 * x619)),
                  (x1170 - (x1171 * x622)), (x1172 - (x1173 * x625)), (x1174 - (x1175 * x628)),
                  (x1176 - (x1177 * x631)), (x1178 - (x1179 * x634)), (x1180 - (x1181 * x637)),
                  (x1182 - (x1183 * x640)), (x1184 - (x1185 * x643)), (x1186 - (x1187 * x646)),
                  (x1188 - (x1189 * x649)), (x1190 - (x1191 * x652)), (x1192 - (x1193 * x655)),
                  (x1194 - (x1195 * x658)), (x1196 - (x1197 * x661)), (x1198 - (x1199 * x664)),
                  (x1200 - (x1201 * x667)), (x1202 - (x1203 * x670)), (x1204 - (x1205 * x673)),
                  (x1206 - (x1207 * x676)), (x1208 - (x1209 * x679)), (x1210 - (x1211 * x682)),
                  (x1532 - (x1533 * x805)), (x1534 - (x1535 * x808)), (x1536 - (x1537 * x811)),
                  (x1538 - (x1539 * x814)), (x1540 - (x1541 * x817)), (x1542 - (x1543 * x820)),
                  (x1544 - (x1545 * x823)), (x1546 - (x1547 * x826)), (x1548 - (x1549 * x829)),
                  (x1550 - (x1551 * x832)), (x1552 - (x1553 * x835)), (x1554 - (x1555 * x838)),
                  (x1556 - (x1557 * x841)), (x1558 - (x1559 * x844)), (x1560 - (x1561 * x847)),
                  (x1562 - (x1563 * x850)), (x1564 - (x1565 * x853)), (x1566 - (x1567 * x856)),
                  (x1568 - (x1569 * x859)), (x1570 - (x1571 * x862)), (x1572 - (x1573 * x865)),
                  (x1574 - (x1575 * x868)), (x1576 - (x1577 * x871)), (x1578 - (x1579 * x874)),
                  (x1580 - (x1581 * x877)), (x1582 - (x1583 * x880)), (x1584 - (x1585 * x883)),
                  (x1586 - (x1587 * x886)), (x1588 - (x1589 * x889)), (x1590 - (x1591 * x892)),
                  (x1592 - (x1593 * x895)), (x1594 - (x1595 * x898)), (x1596 - (x1597 * x94)),
                  (x1598 - (x1599 * x97)),  (x1600 - (x1601 * x100)), (x1602 - (x1603 * x103)),
                  (x1604 - (x1605 * x106)), (x1606 - (x1607 * x109)), (x1608 - (x1609 * x112)),
                  (x1610 - (x1611 * x115)), (x1612 - (x1613 * x118)), (x1614 - (x1615 * x121)),
                  (x1616 - (x1617 * x124)), (x1618 - (x1619 * x127)), (x1620 - (x1621 * x130)),
                  (x1622 - (x1623 * x133)), (x1624 - (x1625 * x136)), (x1626 - (x1627 * x139)),
                  (x1628 - (x1629 * x142)), (x1630 - (x1631 * x145)), (x1632 - (x1633 * x148)),
                  (x1634 - (x1635 * x151)), (x1636 - (x1637 * x154)), (x1638 - (x1639 * x157)),
                  (x1640 - (x1641 * x160)), (x1642 - (x1643 * x163)), (x1644 - (x1645 * x166)),
                  (x1646 - (x1647 * x169)), (x1648 - (x1649 * x172)), (x1650 - (x1651 * x175)),
                  (x1652 - (x1653 * x178)), (x1654 - (x1655 * x181)), (x1656 - (x1657 * x184)),
                  (x1658 - (x1659 * x187)), (x2018 - (x2019 * x346)), (x2020 - (x2021 * x349)),
                  (x2022 - (x2023 * x352)), (x2024 - (x2025 * x355)), (x2026 - (x2027 * x358)),
                  (x2028 - (x2029 * x361)), (x2030 - (x2031 * x364)), (x2032 - (x2033 * x367)),
                  (x2034 - (x2035 * x370)), (x2036 - (x2037 * x373)), (x2038 - (x2039 * x376)),
                  (x2040 - (x2041 * x379)), (x2042 - (x2043 * x382)), (x1980 - (x1981 * x199)),
                  (x1982 - (x1983 * x202)), (x1984 - (x1985 * x205)), (x1986 - (x1987 * x208)),
                  (x1988 - (x1989 * x211)), (x1990 - (x1991 * x214)), (x1992 - (x1993 * x217)),
                  (x1994 - (x1995 * x220)), (x1996 - (x1997 * x223)), (x1998 - (x1999 * x226)),
                  (x2000 - (x2001 * x229)), (x2002 - (x2003 * x232)), (x2004 - (x2005 * x235)),
                  (x2006 - (x2007 * x238)), (x2008 - (x2009 * x241)), (x2010 - (x2011 * x244)),
                  (x2012 - (x2013 * x247)), (x2014 - (x2015 * x250)), (x2016 - (x2017 * x253)),
                  (x2370 - (x2371 * x490)), (x2372 - (x2373 * x493)), (x2374 - (x2375 * x496)),
                  (x2376 - (x2377 * x499)), (x2378 - (x2379 * x502)), (x2380 - (x2381 * x505)),
                  (x2382 - (x2383 * x508)), (x2384 - (x2385 * x511)), (x2386 - (x2387 * x514)),
                  (x2388 - (x2389 * x517)), (x2390 - (x2391 * x520)), (x2392 - (x2393 * x523)),
                  (x2394 - (x2395 * x526)), (x2396 - (x2397 * x529)), (x2398 - (x2399 * x532)),
                  (x2400 - (x2401 * x535)), (x2402 - (x2403 * x538)), (x2404 - (x2405 * x541)),
                  (x2406 - (x2407 * x544)), (x2408 - (x2409 * x547)), (x2410 - (x2411 * x550)),
                  (x2412 - (x2413 * x553)), (x2414 - (x2415 * x556)), (x2416 - (x2417 * x559)),
                  (x2418 - (x2419 * x562)), (x2420 - (x2421 * x565)), (x2422 - (x2423 * x568)),
                  (x2424 - (x2425 * x571)), (x2426 - (x2427 * x574)), (x2364 - (x2365 * x385)),
                  (x2366 - (x2367 * x388)), (x2368 - (x2369 * x391)), (x1020 - (x1021 * x286)),
                  (x1022 - (x1023 * x289)), (x1024 - (x1025 * x292)), (x1026 - (x1027 * x295)),
                  (x1028 - (x1029 * x298)), (x1030 - (x1031 * x301)), (x1032 - (x1033 * x304)),
                  (x1034 - (x1035 * x307)), (x1036 - (x1037 * x310)), (x1038 - (x1039 * x313)),
                  (x1040 - (x1041 * x316)), (x1042 - (x1043 * x319)), (x1044 - (x1045 * x322)),
                  (x1046 - (x1047 * x325)), (x1048 - (x1049 * x328)), (x1050 - (x1051 * x331)),
                  (x1052 - (x1053 * x334)), (x1054 - (x1055 * x337)), (x1056 - (x1057 * x340)),
                  (x1058 - (x1059 * x343)), (x1060 - (x1061 * x346)), (x1062 - (x1063 * x349)),
                  (x1064 - (x1065 * x352)), (x1066 - (x1067 * x355)), (x1068 - (x1069 * x358)),
                  (x1070 - (x1071 * x361)), (x1072 - (x1073 * x364)), (x1074 - (x1075 * x367)),
                  (x1076 - (x1077 * x370)), (x1078 - (x1079 * x373)), (x1080 - (x1081 * x376)),
                  (x1082 - (x1083 * x379)), (x1404 - (x1405 * x463)), (x1406 - (x1407 * x466)),
                  (x1408 - (x1409 * x469)), (x1410 - (x1411 * x472)), (x1412 - (x1413 * x475)),
                  (x1414 - (x1415 * x478)), (x1416 - (x1417 * x481)), (x1418 - (x1419 * x484)),
                  (x1420 - (x1421 * x487)), (x1422 - (x1423 * x490)), (x1424 - (x1425 * x493)),
                  (x1426 - (x1427 * x496)), (x1428 - (x1429 * x499)), (x1430 - (x1431 * x502)),
                  (x1432 - (x1433 * x505)), (x1434 - (x1435 * x508)), (x1436 - (x1437 * x511)),
                  (x1438 - (x1439 * x514)), (x1440 - (x1441 * x517)), (x1442 - (x1443 * x520)),
                  (x1444 - (x1445 * x523)), (x1446 - (x1447 * x526)), (x1448 - (x1449 * x529)),
                  (x1450 - (x1451 * x532)), (x1452 - (x1453 * x535)), (x1454 - (x1455 * x538)),
                  (x1456 - (x1457 * x541)), (x1458 - (x1459 * x544)), (x1460 - (x1461 * x547)),
                  (x1462 - (x1463 * x550)), (x1464 - (x1465 * x553)), (x1466 - (x1467 * x556)),
                  (x1788 - (x1789 * x598)), (x1790 - (x1791 * x601)), (x1792 - (x1793 * x604)),
                  (x1794 - (x1795 * x607)), (x1796 - (x1797 * x610)), (x1798 - (x1799 * x613)),
                  (x1800 - (x1801 * x616)), (x1802 - (x1803 * x619)), (x1804 - (x1805 * x622)),
                  (x1806 - (x1807 * x625)), (x1808 - (x1809 * x628)), (x1810 - (x1811 * x631)),
                  (x1812 - (x1813 * x634)), (x1814 - (x1815 * x637)), (x1816 - (x1817 * x640)),
                  (x1818 - (x1819 * x643)), (x1820 - (x1821 * x646)), (x1822 - (x1823 * x649)),
                  (x1824 - (x1825 * x652)), (x1826 - (x1827 * x655)), (x1828 - (x1829 * x658)),
                  (x1830 - (x1831 * x661)), (x1832 - (x1833 * x664)), (x1834 - (x1835 * x667)),
                  (x1836 - (x1837 * x670)), (x1838 - (x1839 * x673)), (x1840 - (x1841 * x676)),
                  (x1842 - (x1843 * x679)), (x1844 - (x1845 * x682)), (x1846 - (x1847 * x685)),
                  (x1848 - (x1849 * x688)), (x1850 - (x1851 * x691)), (x2172 - (x2173 * x841)),
                  (x2174 - (x2175 * x844)), (x2176 - (x2177 * x847)), (x2178 - (x2179 * x850)),
                  (x2180 - (x2181 * x853)), (x2182 - (x2183 * x856)), (x2184 - (x2185 * x859)),
                  (x2186 - (x2187 * x862)), (x2188 - (x2189 * x865)), (x2190 - (x2191 * x868)),
                  (x2192 - (x2193 * x871)), (x2194 - (x2195 * x874)), (x2196 - (x2197 * x877)),
                  (x2198 - (x2199 * x880)), (x2200 - (x2201 * x883)), (x2202 - (x2203 * x886)),
                  (x2204 - (x2205 * x889)), (x2206 - (x2207 * x892)), (x2208 - (x2209 * x895)),
                  (x2210 - (x2211 * x898)), (x2212 - (x2213 * x901)), (x2214 - (x2215 * x904)),
                  (x2216 - (x2217 * x907)), (x2218 - (x2219 * x910)), (x2220 - (x2221 * x913)),
                  (x2222 - (x2223 * x916)), (x2224 - (x2225 * x919)), (x2226 - (x2227 * x922)),
                  (x2228 - (x2229 * x925)), (x2230 - (x2231 * x928)), (x2232 - (x2233 * x931)),
                  (x2234 - (x2235 * x934)), (x2236 - (x2237 * x49)),  (x2238 - (x2239 * x52)),
                  (x2240 - (x2241 * x55)),  (x2242 - (x2243 * x58)),  (x2244 - (x2245 * x61)),
                  (x2246 - (x2247 * x64)),  (x2248 - (x2249 * x67)),  (x2250 - (x2251 * x70)),
                  (x2252 - (x2253 * x73)),  (x2254 - (x2255 * x76)),  (x2256 - (x2257 * x79)),
                  (x2258 - (x2259 * x82)),  (x2260 - (x2261 * x85)),  (x2262 - (x2263 * x88)),
                  (x2264 - (x2265 * x91)),  (x2266 - (x2267 * x94)),  (x2268 - (x2269 * x97)),
                  (x2270 - (x2271 * x100)), (x2272 - (x2273 * x103)), (x2274 - (x2275 * x106)),
                  (x2276 - (x2277 * x109)), (x2278 - (x2279 * x112)), (x2280 - (x2281 * x115)),
                  (x2282 - (x2283 * x118)), (x2284 - (x2285 * x121)), (x2286 - (x2287 * x124)),
                  (x2288 - (x2289 * x127)), (x2290 - (x2291 * x130)), (x2292 - (x2293 * x133)),
                  (x2294 - (x2295 * x136)), (x2296 - (x2297 * x139)), (x2298 - (x2299 * x142)),
                  (x1212 - (x1213 * x784)), (x1214 - (x1215 * x787)), (x1216 - (x1217 * x790)),
                  (x1218 - (x1219 * x793)), (x1220 - (x1221 * x796)), (x1222 - (x1223 * x799)),
                  (x1224 - (x1225 * x802)), (x1226 - (x1227 * x805)), (x1228 - (x1229 * x808)),
                  (x1230 - (x1231 * x811)), (x1232 - (x1233 * x814)), (x1234 - (x1235 * x817)),
                  (x1236 - (x1237 * x820)), (x1238 - (x1239 * x823)), (x1240 - (x1241 * x826)),
                  (x1242 - (x1243 * x829)), (x1244 - (x1245 * x832)), (x1246 - (x1247 * x835)),
                  (x1248 - (x1249 * x838)), (x1250 - (x1251 * x841)), (x1252 - (x1253 * x844)),
                  (x1254 - (x1255 * x847)), (x1256 - (x1257 * x850)), (x1258 - (x1259 * x853)),
                  (x1260 - (x1261 * x856)), (x1262 - (x1263 * x859)), (x1264 - (x1265 * x862)),
                  (x1266 - (x1267 * x865)), (x1268 - (x1269 * x868)), (x1270 - (x1271 * x871)),
                  (x1272 - (x1273 * x874)), (x1274 - (x1275 * x877)), (x1332 - (x1333 * x187)),
                  (x1334 - (x1335 * x190)), (x1336 - (x1337 * x193)), (x1338 - (x1339 * x196)),
                  (x1276 - (x1277 * x7)),   (x1278 - (x1279 * x10)),  (x1280 - (x1281 * x13)),
                  (x1282 - (x1283 * x16)),  (x1284 - (x1285 * x19)),  (x1286 - (x1287 * x22)),
                  (x1288 - (x1289 * x25)),  (x1290 - (x1291 * x28)),  (x1292 - (x1293 * x31)),
                  (x1294 - (x1295 * x34)),  (x1296 - (x1297 * x37)),  (x1298 - (x1299 * x40)),
                  (x1300 - (x1301 * x43)),  (x1302 - (x1303 * x46)),  (x1304 - (x1305 * x49)),
                  (x1306 - (x1307 * x52)),  (x1308 - (x1309 * x55)),  (x1310 - (x1311 * x58)),
                  (x1312 - (x1313 * x61)),  (x1314 - (x1315 * x64)),  (x1316 - (x1317 * x67)),
                  (x1318 - (x1319 * x70)),  (x1320 - (x1321 * x73)),  (x1322 - (x1323 * x76)),
                  (x1324 - (x1325 * x79)),  (x1326 - (x1327 * x82)),  (x1328 - (x1329 * x85)),
                  (x1330 - (x1331 * x88)),  (x1660 - (x1661 * x259)), (x1662 - (x1663 * x262)),
                  (x1664 - (x1665 * x265)), (x1666 - (x1667 * x268)), (x1668 - (x1669 * x271)),
                  (x1670 - (x1671 * x274)), (x1672 - (x1673 * x277)), (x1674 - (x1675 * x280)),
                  (x1676 - (x1677 * x283)), (x1678 - (x1679 * x286)), (x1680 - (x1681 * x289)),
                  (x1682 - (x1683 * x292)), (x1684 - (x1685 * x295)), (x1686 - (x1687 * x298)),
                  (x1688 - (x1689 * x301)), (x1690 - (x1691 * x304)), (x1692 - (x1693 * x307)),
                  (x1694 - (x1695 * x310)), (x1696 - (x1697 * x313)), (x1698 - (x1699 * x316)),
                  (x1700 - (x1701 * x319)), (x1702 - (x1703 * x322)), (x1704 - (x1705 * x325)),
                  (x1706 - (x1707 * x328)), (x1708 - (x1709 * x331)), (x1710 - (x1711 * x334)),
                  (x1712 - (x1713 * x337)), (x1714 - (x1715 * x340)), (x1716 - (x1717 * x343)),
                  (x1718 - (x1719 * x346)), (x1720 - (x1721 * x349)), (x1722 - (x1723 * x352)),
                  (x2044 - (x2045 * x436)), (x2046 - (x2047 * x439)), (x2048 - (x2049 * x442)),
                  (x2050 - (x2051 * x445)), (x2052 - (x2053 * x448)), (x2054 - (x2055 * x451)),
                  (x2056 - (x2057 * x454)), (x2058 - (x2059 * x457)), (x2060 - (x2061 * x460)),
                  (x2062 - (x2063 * x463)), (x2064 - (x2065 * x466)), (x2066 - (x2067 * x469)),
                  (x2068 - (x2069 * x472)), (x2070 - (x2071 * x475)), (x2072 - (x2073 * x478)),
                  (x2074 - (x2075 * x481)), (x2076 - (x2077 * x484)), (x2078 - (x2079 * x487)),
                  (x2080 - (x2081 * x490)), (x2082 - (x2083 * x493)), (x2084 - (x2085 * x496)),
                  (x2086 - (x2087 * x499)), (x2088 - (x2089 * x502)), (x2090 - (x2091 * x505)),
                  (x2092 - (x2093 * x508)), (x2094 - (x2095 * x511)), (x2096 - (x2097 * x514)),
                  (x2098 - (x2099 * x517)), (x2100 - (x2101 * x520)), (x2102 - (x2103 * x523)),
                  (x2104 - (x2105 * x526)), (x2106 - (x2107 * x529)), (x2444 - (x2445 * x697)),
                  (x2446 - (x2447 * x700)), (x2448 - (x2449 * x703)), (x2450 - (x2451 * x706)),
                  (x2452 - (x2453 * x709)), (x2454 - (x2455 * x712)), (x2456 - (x2457 * x715)),
                  (x2458 - (x2459 * x718)), (x2460 - (x2461 * x721)), (x2462 - (x2463 * x724)),
                  (x2464 - (x2465 * x727)), (x2466 - (x2467 * x730)), (x2468 - (x2469 * x733)),
                  (x2470 - (x2471 * x736)), (x2472 - (x2473 * x739)), (x2474 - (x2475 * x742)),
                  (x2476 - (x2477 * x745)), (x2478 - (x2479 * x748)), (x2480 - (x2481 * x751)),
                  (x2482 - (x2483 * x754)), (x2484 - (x2485 * x757)), (x2486 - (x2487 * x760)),
                  (x2488 - (x2489 * x763)), (x2490 - (x2491 * x766)), (x2428 - (x2429 * x577)),
                  (x2430 - (x2431 * x580)), (x2432 - (x2433 * x583)), (x2434 - (x2435 * x586)),
                  (x2436 - (x2437 * x589)), (x2438 - (x2439 * x592)), (x2440 - (x2441 * x595)),
                  (x2442 - (x2443 * x598)), (x1088 - (x1089 * x487)), (x1090 - (x1091 * x490)),
                  (x1092 - (x1093 * x493)), (x1094 - (x1095 * x496)), (x1096 - (x1097 * x499)),
                  (x1098 - (x1099 * x502)), (x1100 - (x1101 * x505)), (x1102 - (x1103 * x508)),
                  (x1104 - (x1105 * x511)), (x1106 - (x1107 * x514)), (x1108 - (x1109 * x517)),
                  (x1110 - (x1111 * x520)), (x1112 - (x1113 * x523)), (x1114 - (x1115 * x526)),
                  (x1116 - (x1117 * x529)), (x1118 - (x1119 * x532)), (x1120 - (x1121 * x535)),
                  (x1122 - (x1123 * x538)), (x1124 - (x1125 * x541)), (x1126 - (x1127 * x544)),
                  (x1128 - (x1129 * x547)), (x1130 - (x1131 * x550)), (x1132 - (x1133 * x553)),
                  (x1134 - (x1135 * x556)), (x1136 - (x1137 * x559)), (x1138 - (x1139 * x562)),
                  (x1140 - (x1141 * x565)), (x1142 - (x1143 * x568)), (x1144 - (x1145 * x571)),
                  (x1146 - (x1147 * x574)), (x1084 - (x1085 * x385)), (x1086 - (x1087 * x388)),
                  (x1486 - (x1487 * x700)), (x1488 - (x1489 * x703)), (x1490 - (x1491 * x706)),
                  (x1492 - (x1493 * x709)), (x1494 - (x1495 * x712)), (x1496 - (x1497 * x715)),
                  (x1498 - (x1499 * x718)), (x1500 - (x1501 * x721)), (x1502 - (x1503 * x724)),
                  (x1504 - (x1505 * x727)), (x1506 - (x1507 * x730)), (x1508 - (x1509 * x733)),
                  (x1510 - (x1511 * x736)), (x1512 - (x1513 * x739)), (x1514 - (x1515 * x742)),
                  (x1516 - (x1517 * x745)), (x1518 - (x1519 * x748)), (x1520 - (x1521 * x751)),
                  (x1522 - (x1523 * x754)), (x1524 - (x1525 * x757)), (x1526 - (x1527 * x760)),
                  (x1528 - (x1529 * x763)), (x1530 - (x1531 * x766)), (x1468 - (x1469 * x577)),
                  (x1470 - (x1471 * x580)), (x1472 - (x1473 * x583)), (x1474 - (x1475 * x586)),
                  (x1476 - (x1477 * x589)), (x1478 - (x1479 * x592)), (x1480 - (x1481 * x595)),
                  (x1482 - (x1483 * x598)), (x1484 - (x1485 * x601)), (x1902 - (x1903 * x937)),
                  (x1904 - (x1905 * x940)), (x1906 - (x1907 * x943)), (x1908 - (x1909 * x946)),
                  (x1910 - (x1911 * x949)), (x1912 - (x1913 * x952)), (x1914 - (x1915 * x955)),
                  (x1852 - (x1853 * x769)), (x1854 - (x1855 * x772)), (x1856 - (x1857 * x775)),
                  (x1858 - (x1859 * x778)), (x1860 - (x1861 * x781)), (x1862 - (x1863 * x784)),
                  (x1864 - (x1865 * x787)), (x1866 - (x1867 * x790)), (x1868 - (x1869 * x793)),
                  (x1870 - (x1871 * x796)), (x1872 - (x1873 * x799)), (x1874 - (x1875 * x802)),
                  (x1876 - (x1877 * x805)), (x1878 - (x1879 * x808)), (x1880 - (x1881 * x811)),
                  (x1882 - (x1883 * x814)), (x1884 - (x1885 * x817)), (x1886 - (x1887 * x820)),
                  (x1888 - (x1889 * x823)), (x1890 - (x1891 * x826)), (x1892 - (x1893 * x829)),
                  (x1894 - (x1895 * x832)), (x1896 - (x1897 * x835)), (x1898 - (x1899 * x838)),
                  (x1900 - (x1901 * x841)), (x1962 - (x1963 * x172)), (x1964 - (x1965 * x175)),
                  (x1966 - (x1967 * x178)), (x1968 - (x1969 * x181)), (x1970 - (x1971 * x184)),
                  (x1972 - (x1973 * x187)), (x1974 - (x1975 * x190)), (x1976 - (x1977 * x193)),
                  (x1978 - (x1979 * x196)), (x1916 - (x1917 * x7)),   (x1918 - (x1919 * x10)),
                  (x1920 - (x1921 * x13)),  (x1922 - (x1923 * x16)),  (x1924 - (x1925 * x19)),
                  (x1926 - (x1927 * x22)),  (x1928 - (x1929 * x25)),  (x1930 - (x1931 * x28)),
                  (x1932 - (x1933 * x31)),  (x1934 - (x1935 * x34)),  (x1936 - (x1937 * x37)),
                  (x1938 - (x1939 * x40)),  (x1940 - (x1941 * x43)),  (x1942 - (x1943 * x46)),
                  (x1944 - (x1945 * x49)),  (x1946 - (x1947 * x52)),  (x1948 - (x1949 * x55)),
                  (x1950 - (x1951 * x58)),  (x1952 - (x1953 * x61)),  (x1954 - (x1955 * x64)),
                  (x1956 - (x1957 * x67)),  (x1958 - (x1959 * x70)),  (x1960 - (x1961 * x73)),
                  (x2300 - (x2301 * x283)), (x2302 - (x2303 * x286)), (x2304 - (x2305 * x289)),
                  (x2306 - (x2307 * x292)), (x2308 - (x2309 * x295)), (x2310 - (x2311 * x298)),
                  (x2312 - (x2313 * x301)), (x2314 - (x2315 * x304)), (x2316 - (x2317 * x307)),
                  (x2318 - (x2319 * x310)), (x2320 - (x2321 * x313)), (x2322 - (x2323 * x316)),
                  (x2324 - (x2325 * x319)), (x2326 - (x2327 * x322)), (x2328 - (x2329 * x325)),
                  (x2330 - (x2331 * x328)), (x2332 - (x2333 * x331)), (x2334 - (x2335 * x334)),
                  (x2336 - (x2337 * x337)), (x2338 - (x2339 * x340)), (x2340 - (x2341 * x343)),
                  (x2342 - (x2343 * x346)), (x2344 - (x2345 * x349)), (x2346 - (x2347 * x352)),
                  (x2348 - (x2349 * x355)), (x2350 - (x2351 * x358)), (x2352 - (x2353 * x361)),
                  (x2354 - (x2355 * x364)), (x2356 - (x2357 * x367)), (x2358 - (x2359 * x370)),
                  (x2360 - (x2361 * x373)), (x2362 - (x2363 * x376))},
      x2556,
      x2557,
      layout4);
  return x2558;
>>>>>>> origin/main
}

} // namespace risc0::circuit::keccak::cuda
