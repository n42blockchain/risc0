#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "steps.cuh"
#include "witgen.h"

namespace risc0::circuit::keccak::cuda {

<<<<<<< HEAD
__device__ void step_Top_3(ExecContext& ctx, MutableBuf arg0) {
  // ControlState(zirgen/circuit/keccak2/top.zir:403)
  // ComputeCurrentStep(zirgen/circuit/keccak2/top.zir:452)
  // Top(zirgen/circuit/keccak2/top.zir:483)
  set(ctx, arg0, 12, Val(4));
  // ControlState(zirgen/circuit/keccak2/top.zir:404)
  set(ctx, arg0, 13, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:405)
  set(ctx, arg0, 14, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:406)
  set(ctx, arg0, 15, Val(0));
  return;
}
__device__ void step_Top_19(ExecContext& ctx, MutableBuf arg0) {
  // ControlState(zirgen/circuit/keccak2/top.zir:403)
  // ShaNextBlock(zirgen/circuit/keccak2/top.zir:439)
  // ComputeCurrentStep(zirgen/circuit/keccak2/top.zir:463)
  set(ctx, arg0, 12, Val(1));
  // ControlState(zirgen/circuit/keccak2/top.zir:404)
  set(ctx, arg0, 13, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:405)
  set(ctx, arg0, 14, Val(0));
  // ControlState(zirgen/circuit/keccak2/top.zir:406)
  set(ctx, arg0, 15, Val(0));
  return;
}
__device__ void step_Top_35(ExecContext& ctx, MutableBuf arg0) {
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:238)
  // Top(zirgen/circuit/keccak2/top.zir:502)
  Val x1 = (get(ctx, arg0, 240, 1) + (get(ctx, arg0, 241, 1) * Val(2)));
  Val x2 = ((x1 + (get(ctx, arg0, 242, 1) * Val(4))) + (get(ctx, arg0, 243, 1) * Val(8)));
  Val x3 = ((x2 + (get(ctx, arg0, 244, 1) * Val(16))) + (get(ctx, arg0, 245, 1) * Val(32)));
  Val x4 = ((x3 + (get(ctx, arg0, 246, 1) * Val(64))) + (get(ctx, arg0, 247, 1) * Val(128)));
  Val x5 = ((x4 + (get(ctx, arg0, 248, 1) * Val(256))) + (get(ctx, arg0, 249, 1) * Val(512)));
  Val x6 = ((x5 + (get(ctx, arg0, 250, 1) * Val(1024))) + (get(ctx, arg0, 251, 1) * Val(2048)));
  Val x7 = ((x6 + (get(ctx, arg0, 252, 1) * Val(4096))) + (get(ctx, arg0, 253, 1) * Val(8192)));
  Val x8 = ((x7 + (get(ctx, arg0, 254, 1) * Val(16384))) + (get(ctx, arg0, 255, 1) * Val(32768)));
  Val x9 = (get(ctx, arg0, 256, 1) + (get(ctx, arg0, 257, 1) * Val(2)));
  Val x10 = ((x9 + (get(ctx, arg0, 258, 1) * Val(4))) + (get(ctx, arg0, 259, 1) * Val(8)));
  Val x11 = ((x10 + (get(ctx, arg0, 260, 1) * Val(16))) + (get(ctx, arg0, 261, 1) * Val(32)));
  Val x12 = ((x11 + (get(ctx, arg0, 262, 1) * Val(64))) + (get(ctx, arg0, 263, 1) * Val(128)));
  Val x13 = ((x12 + (get(ctx, arg0, 264, 1) * Val(256))) + (get(ctx, arg0, 265, 1) * Val(512)));
  Val x14 = ((x13 + (get(ctx, arg0, 266, 1) * Val(1024))) + (get(ctx, arg0, 267, 1) * Val(2048)));
  Val x15 = ((x14 + (get(ctx, arg0, 268, 1) * Val(4096))) + (get(ctx, arg0, 269, 1) * Val(8192)));
  Val x16 = ((x15 + (get(ctx, arg0, 270, 1) * Val(16384))) + (get(ctx, arg0, 271, 1) * Val(32768)));
  Val x17 = (get(ctx, arg0, 208, 1) + (get(ctx, arg0, 209, 1) * Val(2)));
  Val x18 = ((x17 + (get(ctx, arg0, 210, 1) * Val(4))) + (get(ctx, arg0, 211, 1) * Val(8)));
  Val x19 = ((x18 + (get(ctx, arg0, 212, 1) * Val(16))) + (get(ctx, arg0, 213, 1) * Val(32)));
  Val x20 = ((x19 + (get(ctx, arg0, 214, 1) * Val(64))) + (get(ctx, arg0, 215, 1) * Val(128)));
  Val x21 = ((x20 + (get(ctx, arg0, 216, 1) * Val(256))) + (get(ctx, arg0, 217, 1) * Val(512)));
  Val x22 = ((x21 + (get(ctx, arg0, 218, 1) * Val(1024))) + (get(ctx, arg0, 219, 1) * Val(2048)));
  Val x23 = ((x22 + (get(ctx, arg0, 220, 1) * Val(4096))) + (get(ctx, arg0, 221, 1) * Val(8192)));
  Val x24 = ((x23 + (get(ctx, arg0, 222, 1) * Val(16384))) + (get(ctx, arg0, 223, 1) * Val(32768)));
  Val x25 = (get(ctx, arg0, 224, 1) + (get(ctx, arg0, 225, 1) * Val(2)));
  Val x26 = ((x25 + (get(ctx, arg0, 226, 1) * Val(4))) + (get(ctx, arg0, 227, 1) * Val(8)));
  Val x27 = ((x26 + (get(ctx, arg0, 228, 1) * Val(16))) + (get(ctx, arg0, 229, 1) * Val(32)));
  Val x28 = ((x27 + (get(ctx, arg0, 230, 1) * Val(64))) + (get(ctx, arg0, 231, 1) * Val(128)));
  Val x29 = ((x28 + (get(ctx, arg0, 232, 1) * Val(256))) + (get(ctx, arg0, 233, 1) * Val(512)));
  Val x30 = ((x29 + (get(ctx, arg0, 234, 1) * Val(1024))) + (get(ctx, arg0, 235, 1) * Val(2048)));
  Val x31 = ((x30 + (get(ctx, arg0, 236, 1) * Val(4096))) + (get(ctx, arg0, 237, 1) * Val(8192)));
  Val x32 = ((x31 + (get(ctx, arg0, 238, 1) * Val(16384))) + (get(ctx, arg0, 239, 1) * Val(32768)));
  Val x33 = (get(ctx, arg0, 176, 1) + (get(ctx, arg0, 177, 1) * Val(2)));
  Val x34 = ((x33 + (get(ctx, arg0, 178, 1) * Val(4))) + (get(ctx, arg0, 179, 1) * Val(8)));
  Val x35 = ((x34 + (get(ctx, arg0, 180, 1) * Val(16))) + (get(ctx, arg0, 181, 1) * Val(32)));
  Val x36 = ((x35 + (get(ctx, arg0, 182, 1) * Val(64))) + (get(ctx, arg0, 183, 1) * Val(128)));
  Val x37 = ((x36 + (get(ctx, arg0, 184, 1) * Val(256))) + (get(ctx, arg0, 185, 1) * Val(512)));
  Val x38 = ((x37 + (get(ctx, arg0, 186, 1) * Val(1024))) + (get(ctx, arg0, 187, 1) * Val(2048)));
  Val x39 = ((x38 + (get(ctx, arg0, 188, 1) * Val(4096))) + (get(ctx, arg0, 189, 1) * Val(8192)));
  Val x40 = ((x39 + (get(ctx, arg0, 190, 1) * Val(16384))) + (get(ctx, arg0, 191, 1) * Val(32768)));
  Val x41 = (get(ctx, arg0, 192, 1) + (get(ctx, arg0, 193, 1) * Val(2)));
  Val x42 = ((x41 + (get(ctx, arg0, 194, 1) * Val(4))) + (get(ctx, arg0, 195, 1) * Val(8)));
  Val x43 = ((x42 + (get(ctx, arg0, 196, 1) * Val(16))) + (get(ctx, arg0, 197, 1) * Val(32)));
  Val x44 = ((x43 + (get(ctx, arg0, 198, 1) * Val(64))) + (get(ctx, arg0, 199, 1) * Val(128)));
  Val x45 = ((x44 + (get(ctx, arg0, 200, 1) * Val(256))) + (get(ctx, arg0, 201, 1) * Val(512)));
  Val x46 = ((x45 + (get(ctx, arg0, 202, 1) * Val(1024))) + (get(ctx, arg0, 203, 1) * Val(2048)));
  Val x47 = ((x46 + (get(ctx, arg0, 204, 1) * Val(4096))) + (get(ctx, arg0, 205, 1) * Val(8192)));
  Val x48 = ((x47 + (get(ctx, arg0, 206, 1) * Val(16384))) + (get(ctx, arg0, 207, 1) * Val(32768)));
  Val x49 = (get(ctx, arg0, 144, 1) + (get(ctx, arg0, 145, 1) * Val(2)));
  Val x50 = ((x49 + (get(ctx, arg0, 146, 1) * Val(4))) + (get(ctx, arg0, 147, 1) * Val(8)));
  Val x51 = ((x50 + (get(ctx, arg0, 148, 1) * Val(16))) + (get(ctx, arg0, 149, 1) * Val(32)));
  Val x52 = ((x51 + (get(ctx, arg0, 150, 1) * Val(64))) + (get(ctx, arg0, 151, 1) * Val(128)));
  Val x53 = ((x52 + (get(ctx, arg0, 152, 1) * Val(256))) + (get(ctx, arg0, 153, 1) * Val(512)));
  Val x54 = ((x53 + (get(ctx, arg0, 154, 1) * Val(1024))) + (get(ctx, arg0, 155, 1) * Val(2048)));
  Val x55 = ((x54 + (get(ctx, arg0, 156, 1) * Val(4096))) + (get(ctx, arg0, 157, 1) * Val(8192)));
  Val x56 = ((x55 + (get(ctx, arg0, 158, 1) * Val(16384))) + (get(ctx, arg0, 159, 1) * Val(32768)));
  Val x57 = (get(ctx, arg0, 160, 1) + (get(ctx, arg0, 161, 1) * Val(2)));
  Val x58 = ((x57 + (get(ctx, arg0, 162, 1) * Val(4))) + (get(ctx, arg0, 163, 1) * Val(8)));
  Val x59 = ((x58 + (get(ctx, arg0, 164, 1) * Val(16))) + (get(ctx, arg0, 165, 1) * Val(32)));
  Val x60 = ((x59 + (get(ctx, arg0, 166, 1) * Val(64))) + (get(ctx, arg0, 167, 1) * Val(128)));
  Val x61 = ((x60 + (get(ctx, arg0, 168, 1) * Val(256))) + (get(ctx, arg0, 169, 1) * Val(512)));
  Val x62 = ((x61 + (get(ctx, arg0, 170, 1) * Val(1024))) + (get(ctx, arg0, 171, 1) * Val(2048)));
  Val x63 = ((x62 + (get(ctx, arg0, 172, 1) * Val(4096))) + (get(ctx, arg0, 173, 1) * Val(8192)));
  Val x64 = ((x63 + (get(ctx, arg0, 174, 1) * Val(16384))) + (get(ctx, arg0, 175, 1) * Val(32768)));
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:239)
  Val x65 = (get(ctx, arg0, 496, 1) + (get(ctx, arg0, 497, 1) * Val(2)));
  Val x66 = ((x65 + (get(ctx, arg0, 498, 1) * Val(4))) + (get(ctx, arg0, 499, 1) * Val(8)));
  Val x67 = ((x66 + (get(ctx, arg0, 500, 1) * Val(16))) + (get(ctx, arg0, 501, 1) * Val(32)));
  Val x68 = ((x67 + (get(ctx, arg0, 502, 1) * Val(64))) + (get(ctx, arg0, 503, 1) * Val(128)));
  Val x69 = ((x68 + (get(ctx, arg0, 504, 1) * Val(256))) + (get(ctx, arg0, 505, 1) * Val(512)));
  Val x70 = ((x69 + (get(ctx, arg0, 506, 1) * Val(1024))) + (get(ctx, arg0, 507, 1) * Val(2048)));
  Val x71 = ((x70 + (get(ctx, arg0, 508, 1) * Val(4096))) + (get(ctx, arg0, 509, 1) * Val(8192)));
  Val x72 = ((x71 + (get(ctx, arg0, 510, 1) * Val(16384))) + (get(ctx, arg0, 511, 1) * Val(32768)));
  Val x73 = (get(ctx, arg0, 512, 1) + (get(ctx, arg0, 513, 1) * Val(2)));
  Val x74 = ((x73 + (get(ctx, arg0, 514, 1) * Val(4))) + (get(ctx, arg0, 515, 1) * Val(8)));
  Val x75 = ((x74 + (get(ctx, arg0, 516, 1) * Val(16))) + (get(ctx, arg0, 517, 1) * Val(32)));
  Val x76 = ((x75 + (get(ctx, arg0, 518, 1) * Val(64))) + (get(ctx, arg0, 519, 1) * Val(128)));
  Val x77 = ((x76 + (get(ctx, arg0, 520, 1) * Val(256))) + (get(ctx, arg0, 521, 1) * Val(512)));
  Val x78 = ((x77 + (get(ctx, arg0, 522, 1) * Val(1024))) + (get(ctx, arg0, 523, 1) * Val(2048)));
  Val x79 = ((x78 + (get(ctx, arg0, 524, 1) * Val(4096))) + (get(ctx, arg0, 525, 1) * Val(8192)));
  Val x80 = ((x79 + (get(ctx, arg0, 526, 1) * Val(16384))) + (get(ctx, arg0, 527, 1) * Val(32768)));
  Val x81 = (get(ctx, arg0, 464, 1) + (get(ctx, arg0, 465, 1) * Val(2)));
  Val x82 = ((x81 + (get(ctx, arg0, 466, 1) * Val(4))) + (get(ctx, arg0, 467, 1) * Val(8)));
  Val x83 = ((x82 + (get(ctx, arg0, 468, 1) * Val(16))) + (get(ctx, arg0, 469, 1) * Val(32)));
  Val x84 = ((x83 + (get(ctx, arg0, 470, 1) * Val(64))) + (get(ctx, arg0, 471, 1) * Val(128)));
  Val x85 = ((x84 + (get(ctx, arg0, 472, 1) * Val(256))) + (get(ctx, arg0, 473, 1) * Val(512)));
  Val x86 = ((x85 + (get(ctx, arg0, 474, 1) * Val(1024))) + (get(ctx, arg0, 475, 1) * Val(2048)));
  Val x87 = ((x86 + (get(ctx, arg0, 476, 1) * Val(4096))) + (get(ctx, arg0, 477, 1) * Val(8192)));
  Val x88 = ((x87 + (get(ctx, arg0, 478, 1) * Val(16384))) + (get(ctx, arg0, 479, 1) * Val(32768)));
  Val x89 = (get(ctx, arg0, 480, 1) + (get(ctx, arg0, 481, 1) * Val(2)));
  Val x90 = ((x89 + (get(ctx, arg0, 482, 1) * Val(4))) + (get(ctx, arg0, 483, 1) * Val(8)));
  Val x91 = ((x90 + (get(ctx, arg0, 484, 1) * Val(16))) + (get(ctx, arg0, 485, 1) * Val(32)));
  Val x92 = ((x91 + (get(ctx, arg0, 486, 1) * Val(64))) + (get(ctx, arg0, 487, 1) * Val(128)));
  Val x93 = ((x92 + (get(ctx, arg0, 488, 1) * Val(256))) + (get(ctx, arg0, 489, 1) * Val(512)));
  Val x94 = ((x93 + (get(ctx, arg0, 490, 1) * Val(1024))) + (get(ctx, arg0, 491, 1) * Val(2048)));
  Val x95 = ((x94 + (get(ctx, arg0, 492, 1) * Val(4096))) + (get(ctx, arg0, 493, 1) * Val(8192)));
  Val x96 = ((x95 + (get(ctx, arg0, 494, 1) * Val(16384))) + (get(ctx, arg0, 495, 1) * Val(32768)));
  Val x97 = (get(ctx, arg0, 432, 1) + (get(ctx, arg0, 433, 1) * Val(2)));
  Val x98 = ((x97 + (get(ctx, arg0, 434, 1) * Val(4))) + (get(ctx, arg0, 435, 1) * Val(8)));
  Val x99 = ((x98 + (get(ctx, arg0, 436, 1) * Val(16))) + (get(ctx, arg0, 437, 1) * Val(32)));
  Val x100 = ((x99 + (get(ctx, arg0, 438, 1) * Val(64))) + (get(ctx, arg0, 439, 1) * Val(128)));
  Val x101 = ((x100 + (get(ctx, arg0, 440, 1) * Val(256))) + (get(ctx, arg0, 441, 1) * Val(512)));
  Val x102 = ((x101 + (get(ctx, arg0, 442, 1) * Val(1024))) + (get(ctx, arg0, 443, 1) * Val(2048)));
  Val x103 = ((x102 + (get(ctx, arg0, 444, 1) * Val(4096))) + (get(ctx, arg0, 445, 1) * Val(8192)));
  Val x104 =
      ((x103 + (get(ctx, arg0, 446, 1) * Val(16384))) + (get(ctx, arg0, 447, 1) * Val(32768)));
  Val x105 = (get(ctx, arg0, 448, 1) + (get(ctx, arg0, 449, 1) * Val(2)));
  Val x106 = ((x105 + (get(ctx, arg0, 450, 1) * Val(4))) + (get(ctx, arg0, 451, 1) * Val(8)));
  Val x107 = ((x106 + (get(ctx, arg0, 452, 1) * Val(16))) + (get(ctx, arg0, 453, 1) * Val(32)));
  Val x108 = ((x107 + (get(ctx, arg0, 454, 1) * Val(64))) + (get(ctx, arg0, 455, 1) * Val(128)));
  Val x109 = ((x108 + (get(ctx, arg0, 456, 1) * Val(256))) + (get(ctx, arg0, 457, 1) * Val(512)));
  Val x110 = ((x109 + (get(ctx, arg0, 458, 1) * Val(1024))) + (get(ctx, arg0, 459, 1) * Val(2048)));
  Val x111 = ((x110 + (get(ctx, arg0, 460, 1) * Val(4096))) + (get(ctx, arg0, 461, 1) * Val(8192)));
  Val x112 =
      ((x111 + (get(ctx, arg0, 462, 1) * Val(16384))) + (get(ctx, arg0, 463, 1) * Val(32768)));
  Val x113 = (get(ctx, arg0, 400, 1) + (get(ctx, arg0, 401, 1) * Val(2)));
  Val x114 = ((x113 + (get(ctx, arg0, 402, 1) * Val(4))) + (get(ctx, arg0, 403, 1) * Val(8)));
  Val x115 = ((x114 + (get(ctx, arg0, 404, 1) * Val(16))) + (get(ctx, arg0, 405, 1) * Val(32)));
  Val x116 = ((x115 + (get(ctx, arg0, 406, 1) * Val(64))) + (get(ctx, arg0, 407, 1) * Val(128)));
  Val x117 = ((x116 + (get(ctx, arg0, 408, 1) * Val(256))) + (get(ctx, arg0, 409, 1) * Val(512)));
  Val x118 = ((x117 + (get(ctx, arg0, 410, 1) * Val(1024))) + (get(ctx, arg0, 411, 1) * Val(2048)));
  Val x119 = ((x118 + (get(ctx, arg0, 412, 1) * Val(4096))) + (get(ctx, arg0, 413, 1) * Val(8192)));
  Val x120 =
      ((x119 + (get(ctx, arg0, 414, 1) * Val(16384))) + (get(ctx, arg0, 415, 1) * Val(32768)));
  Val x121 = (get(ctx, arg0, 416, 1) + (get(ctx, arg0, 417, 1) * Val(2)));
  Val x122 = ((x121 + (get(ctx, arg0, 418, 1) * Val(4))) + (get(ctx, arg0, 419, 1) * Val(8)));
  Val x123 = ((x122 + (get(ctx, arg0, 420, 1) * Val(16))) + (get(ctx, arg0, 421, 1) * Val(32)));
  Val x124 = ((x123 + (get(ctx, arg0, 422, 1) * Val(64))) + (get(ctx, arg0, 423, 1) * Val(128)));
  Val x125 = ((x124 + (get(ctx, arg0, 424, 1) * Val(256))) + (get(ctx, arg0, 425, 1) * Val(512)));
  Val x126 = ((x125 + (get(ctx, arg0, 426, 1) * Val(1024))) + (get(ctx, arg0, 427, 1) * Val(2048)));
  Val x127 = ((x126 + (get(ctx, arg0, 428, 1) * Val(4096))) + (get(ctx, arg0, 429, 1) * Val(8192)));
  Val x128 =
      ((x127 + (get(ctx, arg0, 430, 1) * Val(16384))) + (get(ctx, arg0, 431, 1) * Val(32768)));
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:244)
  Val x129 = (x8 + get(ctx, arg0, 916, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x130 = (bitAnd(x129, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 937, bitAnd(x130, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 938, (bitAnd(x130, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 939, (bitAnd(x130, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x131 = ((get(ctx, arg0, 939, 0) * Val(4)) + (get(ctx, arg0, 938, 0) * Val(2)));
  Val x132 = (x131 + get(ctx, arg0, 937, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x133 = (x129 - (x132 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x134 = ((x16 + get(ctx, arg0, 917, 1)) + x132);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x135 = (bitAnd(x134, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 940, bitAnd(x135, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 941, (bitAnd(x135, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 942, (bitAnd(x135, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x136 = ((get(ctx, arg0, 942, 0) * Val(4)) + (get(ctx, arg0, 941, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x137 = (x134 - ((x136 + get(ctx, arg0, 940, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 240, bitAnd(x133, Val(1)));
  Val x138 = get(ctx, arg0, 240, 0);
  set(ctx, arg0, 241, (bitAnd(x133, Val(2)) * Val(1006632961)));
  Val x139 = get(ctx, arg0, 241, 0);
  set(ctx, arg0, 242, (bitAnd(x133, Val(4)) * Val(1509949441)));
  Val x140 = get(ctx, arg0, 242, 0);
  set(ctx, arg0, 243, (bitAnd(x133, Val(8)) * Val(1761607681)));
  Val x141 = get(ctx, arg0, 243, 0);
  set(ctx, arg0, 244, (bitAnd(x133, Val(16)) * Val(1887436801)));
  Val x142 = get(ctx, arg0, 244, 0);
  set(ctx, arg0, 245, (bitAnd(x133, Val(32)) * Val(1950351361)));
  Val x143 = get(ctx, arg0, 245, 0);
  set(ctx, arg0, 246, (bitAnd(x133, Val(64)) * Val(1981808641)));
  Val x144 = get(ctx, arg0, 246, 0);
  set(ctx, arg0, 247, (bitAnd(x133, Val(128)) * Val(1997537281)));
  Val x145 = get(ctx, arg0, 247, 0);
  set(ctx, arg0, 248, (bitAnd(x133, Val(256)) * Val(2005401601)));
  Val x146 = get(ctx, arg0, 248, 0);
  set(ctx, arg0, 249, (bitAnd(x133, Val(512)) * Val(2009333761)));
  Val x147 = get(ctx, arg0, 249, 0);
  set(ctx, arg0, 250, (bitAnd(x133, Val(1024)) * Val(2011299841)));
  Val x148 = get(ctx, arg0, 250, 0);
  set(ctx, arg0, 251, (bitAnd(x133, Val(2048)) * Val(2012282881)));
  Val x149 = get(ctx, arg0, 251, 0);
  set(ctx, arg0, 252, (bitAnd(x133, Val(4096)) * Val(2012774401)));
  Val x150 = get(ctx, arg0, 252, 0);
  set(ctx, arg0, 253, (bitAnd(x133, Val(8192)) * Val(2013020161)));
  Val x151 = get(ctx, arg0, 253, 0);
  set(ctx, arg0, 254, (bitAnd(x133, Val(16384)) * Val(2013143041)));
  Val x152 = get(ctx, arg0, 254, 0);
  set(ctx, arg0, 255, (bitAnd(x133, Val(32768)) * Val(2013204481)));
  Val x153 = get(ctx, arg0, 255, 0);
  set(ctx, arg0, 256, bitAnd(x137, Val(1)));
  Val x154 = get(ctx, arg0, 256, 0);
  set(ctx, arg0, 257, (bitAnd(x137, Val(2)) * Val(1006632961)));
  Val x155 = get(ctx, arg0, 257, 0);
  set(ctx, arg0, 258, (bitAnd(x137, Val(4)) * Val(1509949441)));
  Val x156 = get(ctx, arg0, 258, 0);
  set(ctx, arg0, 259, (bitAnd(x137, Val(8)) * Val(1761607681)));
  Val x157 = get(ctx, arg0, 259, 0);
  set(ctx, arg0, 260, (bitAnd(x137, Val(16)) * Val(1887436801)));
  Val x158 = get(ctx, arg0, 260, 0);
  set(ctx, arg0, 261, (bitAnd(x137, Val(32)) * Val(1950351361)));
  Val x159 = get(ctx, arg0, 261, 0);
  set(ctx, arg0, 262, (bitAnd(x137, Val(64)) * Val(1981808641)));
  Val x160 = get(ctx, arg0, 262, 0);
  set(ctx, arg0, 263, (bitAnd(x137, Val(128)) * Val(1997537281)));
  Val x161 = get(ctx, arg0, 263, 0);
  set(ctx, arg0, 264, (bitAnd(x137, Val(256)) * Val(2005401601)));
  Val x162 = get(ctx, arg0, 264, 0);
  set(ctx, arg0, 265, (bitAnd(x137, Val(512)) * Val(2009333761)));
  Val x163 = get(ctx, arg0, 265, 0);
  set(ctx, arg0, 266, (bitAnd(x137, Val(1024)) * Val(2011299841)));
  Val x164 = get(ctx, arg0, 266, 0);
  set(ctx, arg0, 267, (bitAnd(x137, Val(2048)) * Val(2012282881)));
  Val x165 = get(ctx, arg0, 267, 0);
  set(ctx, arg0, 268, (bitAnd(x137, Val(4096)) * Val(2012774401)));
  Val x166 = get(ctx, arg0, 268, 0);
  set(ctx, arg0, 269, (bitAnd(x137, Val(8192)) * Val(2013020161)));
  Val x167 = get(ctx, arg0, 269, 0);
  set(ctx, arg0, 270, (bitAnd(x137, Val(16384)) * Val(2013143041)));
  Val x168 = get(ctx, arg0, 270, 0);
  set(ctx, arg0, 271, (bitAnd(x137, Val(32768)) * Val(2013204481)));
  Val x169 = get(ctx, arg0, 271, 0);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x170 = (x24 + get(ctx, arg0, 918, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x171 = (bitAnd(x170, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 943, bitAnd(x171, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 944, (bitAnd(x171, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 945, (bitAnd(x171, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x172 = ((get(ctx, arg0, 945, 0) * Val(4)) + (get(ctx, arg0, 944, 0) * Val(2)));
  Val x173 = (x172 + get(ctx, arg0, 943, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x174 = (x170 - (x173 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x175 = ((x32 + get(ctx, arg0, 919, 1)) + x173);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x176 = (bitAnd(x175, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 946, bitAnd(x176, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 947, (bitAnd(x176, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 948, (bitAnd(x176, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x177 = ((get(ctx, arg0, 948, 0) * Val(4)) + (get(ctx, arg0, 947, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x178 = (x175 - ((x177 + get(ctx, arg0, 946, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 208, bitAnd(x174, Val(1)));
  Val x179 = get(ctx, arg0, 208, 0);
  set(ctx, arg0, 209, (bitAnd(x174, Val(2)) * Val(1006632961)));
  Val x180 = get(ctx, arg0, 209, 0);
  set(ctx, arg0, 210, (bitAnd(x174, Val(4)) * Val(1509949441)));
  Val x181 = get(ctx, arg0, 210, 0);
  set(ctx, arg0, 211, (bitAnd(x174, Val(8)) * Val(1761607681)));
  Val x182 = get(ctx, arg0, 211, 0);
  set(ctx, arg0, 212, (bitAnd(x174, Val(16)) * Val(1887436801)));
  Val x183 = get(ctx, arg0, 212, 0);
  set(ctx, arg0, 213, (bitAnd(x174, Val(32)) * Val(1950351361)));
  Val x184 = get(ctx, arg0, 213, 0);
  set(ctx, arg0, 214, (bitAnd(x174, Val(64)) * Val(1981808641)));
  Val x185 = get(ctx, arg0, 214, 0);
  set(ctx, arg0, 215, (bitAnd(x174, Val(128)) * Val(1997537281)));
  Val x186 = get(ctx, arg0, 215, 0);
  set(ctx, arg0, 216, (bitAnd(x174, Val(256)) * Val(2005401601)));
  Val x187 = get(ctx, arg0, 216, 0);
  set(ctx, arg0, 217, (bitAnd(x174, Val(512)) * Val(2009333761)));
  Val x188 = get(ctx, arg0, 217, 0);
  set(ctx, arg0, 218, (bitAnd(x174, Val(1024)) * Val(2011299841)));
  Val x189 = get(ctx, arg0, 218, 0);
  set(ctx, arg0, 219, (bitAnd(x174, Val(2048)) * Val(2012282881)));
  Val x190 = get(ctx, arg0, 219, 0);
  set(ctx, arg0, 220, (bitAnd(x174, Val(4096)) * Val(2012774401)));
  Val x191 = get(ctx, arg0, 220, 0);
  set(ctx, arg0, 221, (bitAnd(x174, Val(8192)) * Val(2013020161)));
  Val x192 = get(ctx, arg0, 221, 0);
  set(ctx, arg0, 222, (bitAnd(x174, Val(16384)) * Val(2013143041)));
  Val x193 = get(ctx, arg0, 222, 0);
  set(ctx, arg0, 223, (bitAnd(x174, Val(32768)) * Val(2013204481)));
  Val x194 = get(ctx, arg0, 223, 0);
  set(ctx, arg0, 224, bitAnd(x178, Val(1)));
  Val x195 = get(ctx, arg0, 224, 0);
  set(ctx, arg0, 225, (bitAnd(x178, Val(2)) * Val(1006632961)));
  Val x196 = get(ctx, arg0, 225, 0);
  set(ctx, arg0, 226, (bitAnd(x178, Val(4)) * Val(1509949441)));
  Val x197 = get(ctx, arg0, 226, 0);
  set(ctx, arg0, 227, (bitAnd(x178, Val(8)) * Val(1761607681)));
  Val x198 = get(ctx, arg0, 227, 0);
  set(ctx, arg0, 228, (bitAnd(x178, Val(16)) * Val(1887436801)));
  Val x199 = get(ctx, arg0, 228, 0);
  set(ctx, arg0, 229, (bitAnd(x178, Val(32)) * Val(1950351361)));
  Val x200 = get(ctx, arg0, 229, 0);
  set(ctx, arg0, 230, (bitAnd(x178, Val(64)) * Val(1981808641)));
  Val x201 = get(ctx, arg0, 230, 0);
  set(ctx, arg0, 231, (bitAnd(x178, Val(128)) * Val(1997537281)));
  Val x202 = get(ctx, arg0, 231, 0);
  set(ctx, arg0, 232, (bitAnd(x178, Val(256)) * Val(2005401601)));
  Val x203 = get(ctx, arg0, 232, 0);
  set(ctx, arg0, 233, (bitAnd(x178, Val(512)) * Val(2009333761)));
  Val x204 = get(ctx, arg0, 233, 0);
  set(ctx, arg0, 234, (bitAnd(x178, Val(1024)) * Val(2011299841)));
  Val x205 = get(ctx, arg0, 234, 0);
  set(ctx, arg0, 235, (bitAnd(x178, Val(2048)) * Val(2012282881)));
  Val x206 = get(ctx, arg0, 235, 0);
  set(ctx, arg0, 236, (bitAnd(x178, Val(4096)) * Val(2012774401)));
  Val x207 = get(ctx, arg0, 236, 0);
  set(ctx, arg0, 237, (bitAnd(x178, Val(8192)) * Val(2013020161)));
  Val x208 = get(ctx, arg0, 237, 0);
  set(ctx, arg0, 238, (bitAnd(x178, Val(16384)) * Val(2013143041)));
  Val x209 = get(ctx, arg0, 238, 0);
  set(ctx, arg0, 239, (bitAnd(x178, Val(32768)) * Val(2013204481)));
  Val x210 = get(ctx, arg0, 239, 0);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x211 = (x40 + get(ctx, arg0, 920, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x212 = (bitAnd(x211, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 949, bitAnd(x212, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 950, (bitAnd(x212, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 951, (bitAnd(x212, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x213 = ((get(ctx, arg0, 951, 0) * Val(4)) + (get(ctx, arg0, 950, 0) * Val(2)));
  Val x214 = (x213 + get(ctx, arg0, 949, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x215 = (x211 - (x214 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x216 = ((x48 + get(ctx, arg0, 921, 1)) + x214);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x217 = (bitAnd(x216, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 952, bitAnd(x217, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 953, (bitAnd(x217, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 954, (bitAnd(x217, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x218 = ((get(ctx, arg0, 954, 0) * Val(4)) + (get(ctx, arg0, 953, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x219 = (x216 - ((x218 + get(ctx, arg0, 952, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 176, bitAnd(x215, Val(1)));
  Val x220 = get(ctx, arg0, 176, 0);
  set(ctx, arg0, 177, (bitAnd(x215, Val(2)) * Val(1006632961)));
  Val x221 = get(ctx, arg0, 177, 0);
  set(ctx, arg0, 178, (bitAnd(x215, Val(4)) * Val(1509949441)));
  Val x222 = get(ctx, arg0, 178, 0);
  set(ctx, arg0, 179, (bitAnd(x215, Val(8)) * Val(1761607681)));
  Val x223 = get(ctx, arg0, 179, 0);
  set(ctx, arg0, 180, (bitAnd(x215, Val(16)) * Val(1887436801)));
  Val x224 = get(ctx, arg0, 180, 0);
  set(ctx, arg0, 181, (bitAnd(x215, Val(32)) * Val(1950351361)));
  Val x225 = get(ctx, arg0, 181, 0);
  set(ctx, arg0, 182, (bitAnd(x215, Val(64)) * Val(1981808641)));
  Val x226 = get(ctx, arg0, 182, 0);
  set(ctx, arg0, 183, (bitAnd(x215, Val(128)) * Val(1997537281)));
  Val x227 = get(ctx, arg0, 183, 0);
  set(ctx, arg0, 184, (bitAnd(x215, Val(256)) * Val(2005401601)));
  Val x228 = get(ctx, arg0, 184, 0);
  set(ctx, arg0, 185, (bitAnd(x215, Val(512)) * Val(2009333761)));
  Val x229 = get(ctx, arg0, 185, 0);
  set(ctx, arg0, 186, (bitAnd(x215, Val(1024)) * Val(2011299841)));
  Val x230 = get(ctx, arg0, 186, 0);
  set(ctx, arg0, 187, (bitAnd(x215, Val(2048)) * Val(2012282881)));
  Val x231 = get(ctx, arg0, 187, 0);
  set(ctx, arg0, 188, (bitAnd(x215, Val(4096)) * Val(2012774401)));
  Val x232 = get(ctx, arg0, 188, 0);
  set(ctx, arg0, 189, (bitAnd(x215, Val(8192)) * Val(2013020161)));
  Val x233 = get(ctx, arg0, 189, 0);
  set(ctx, arg0, 190, (bitAnd(x215, Val(16384)) * Val(2013143041)));
  Val x234 = get(ctx, arg0, 190, 0);
  set(ctx, arg0, 191, (bitAnd(x215, Val(32768)) * Val(2013204481)));
  Val x235 = get(ctx, arg0, 191, 0);
  set(ctx, arg0, 192, bitAnd(x219, Val(1)));
  Val x236 = get(ctx, arg0, 192, 0);
  set(ctx, arg0, 193, (bitAnd(x219, Val(2)) * Val(1006632961)));
  Val x237 = get(ctx, arg0, 193, 0);
  set(ctx, arg0, 194, (bitAnd(x219, Val(4)) * Val(1509949441)));
  Val x238 = get(ctx, arg0, 194, 0);
  set(ctx, arg0, 195, (bitAnd(x219, Val(8)) * Val(1761607681)));
  Val x239 = get(ctx, arg0, 195, 0);
  set(ctx, arg0, 196, (bitAnd(x219, Val(16)) * Val(1887436801)));
  Val x240 = get(ctx, arg0, 196, 0);
  set(ctx, arg0, 197, (bitAnd(x219, Val(32)) * Val(1950351361)));
  Val x241 = get(ctx, arg0, 197, 0);
  set(ctx, arg0, 198, (bitAnd(x219, Val(64)) * Val(1981808641)));
  Val x242 = get(ctx, arg0, 198, 0);
  set(ctx, arg0, 199, (bitAnd(x219, Val(128)) * Val(1997537281)));
  Val x243 = get(ctx, arg0, 199, 0);
  set(ctx, arg0, 200, (bitAnd(x219, Val(256)) * Val(2005401601)));
  Val x244 = get(ctx, arg0, 200, 0);
  set(ctx, arg0, 201, (bitAnd(x219, Val(512)) * Val(2009333761)));
  Val x245 = get(ctx, arg0, 201, 0);
  set(ctx, arg0, 202, (bitAnd(x219, Val(1024)) * Val(2011299841)));
  Val x246 = get(ctx, arg0, 202, 0);
  set(ctx, arg0, 203, (bitAnd(x219, Val(2048)) * Val(2012282881)));
  Val x247 = get(ctx, arg0, 203, 0);
  set(ctx, arg0, 204, (bitAnd(x219, Val(4096)) * Val(2012774401)));
  Val x248 = get(ctx, arg0, 204, 0);
  set(ctx, arg0, 205, (bitAnd(x219, Val(8192)) * Val(2013020161)));
  Val x249 = get(ctx, arg0, 205, 0);
  set(ctx, arg0, 206, (bitAnd(x219, Val(16384)) * Val(2013143041)));
  Val x250 = get(ctx, arg0, 206, 0);
  set(ctx, arg0, 207, (bitAnd(x219, Val(32768)) * Val(2013204481)));
  Val x251 = get(ctx, arg0, 207, 0);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x252 = (x56 + get(ctx, arg0, 922, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x253 = (bitAnd(x252, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 955, bitAnd(x253, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 956, (bitAnd(x253, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 957, (bitAnd(x253, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x254 = ((get(ctx, arg0, 957, 0) * Val(4)) + (get(ctx, arg0, 956, 0) * Val(2)));
  Val x255 = (x254 + get(ctx, arg0, 955, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x256 = (x252 - (x255 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x257 = ((x64 + get(ctx, arg0, 923, 1)) + x255);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x258 = (bitAnd(x257, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 958, bitAnd(x258, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 959, (bitAnd(x258, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 960, (bitAnd(x258, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x259 = ((get(ctx, arg0, 960, 0) * Val(4)) + (get(ctx, arg0, 959, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x260 = (x257 - ((x259 + get(ctx, arg0, 958, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 144, bitAnd(x256, Val(1)));
  Val x261 = get(ctx, arg0, 144, 0);
  set(ctx, arg0, 145, (bitAnd(x256, Val(2)) * Val(1006632961)));
  Val x262 = get(ctx, arg0, 145, 0);
  set(ctx, arg0, 146, (bitAnd(x256, Val(4)) * Val(1509949441)));
  Val x263 = get(ctx, arg0, 146, 0);
  set(ctx, arg0, 147, (bitAnd(x256, Val(8)) * Val(1761607681)));
  Val x264 = get(ctx, arg0, 147, 0);
  set(ctx, arg0, 148, (bitAnd(x256, Val(16)) * Val(1887436801)));
  Val x265 = get(ctx, arg0, 148, 0);
  set(ctx, arg0, 149, (bitAnd(x256, Val(32)) * Val(1950351361)));
  Val x266 = get(ctx, arg0, 149, 0);
  set(ctx, arg0, 150, (bitAnd(x256, Val(64)) * Val(1981808641)));
  Val x267 = get(ctx, arg0, 150, 0);
  set(ctx, arg0, 151, (bitAnd(x256, Val(128)) * Val(1997537281)));
  Val x268 = get(ctx, arg0, 151, 0);
  set(ctx, arg0, 152, (bitAnd(x256, Val(256)) * Val(2005401601)));
  Val x269 = get(ctx, arg0, 152, 0);
  set(ctx, arg0, 153, (bitAnd(x256, Val(512)) * Val(2009333761)));
  Val x270 = get(ctx, arg0, 153, 0);
  set(ctx, arg0, 154, (bitAnd(x256, Val(1024)) * Val(2011299841)));
  Val x271 = get(ctx, arg0, 154, 0);
  set(ctx, arg0, 155, (bitAnd(x256, Val(2048)) * Val(2012282881)));
  Val x272 = get(ctx, arg0, 155, 0);
  set(ctx, arg0, 156, (bitAnd(x256, Val(4096)) * Val(2012774401)));
  Val x273 = get(ctx, arg0, 156, 0);
  set(ctx, arg0, 157, (bitAnd(x256, Val(8192)) * Val(2013020161)));
  Val x274 = get(ctx, arg0, 157, 0);
  set(ctx, arg0, 158, (bitAnd(x256, Val(16384)) * Val(2013143041)));
  Val x275 = get(ctx, arg0, 158, 0);
  set(ctx, arg0, 159, (bitAnd(x256, Val(32768)) * Val(2013204481)));
  Val x276 = get(ctx, arg0, 159, 0);
  set(ctx, arg0, 160, bitAnd(x260, Val(1)));
  Val x277 = get(ctx, arg0, 160, 0);
  set(ctx, arg0, 161, (bitAnd(x260, Val(2)) * Val(1006632961)));
  Val x278 = get(ctx, arg0, 161, 0);
  set(ctx, arg0, 162, (bitAnd(x260, Val(4)) * Val(1509949441)));
  Val x279 = get(ctx, arg0, 162, 0);
  set(ctx, arg0, 163, (bitAnd(x260, Val(8)) * Val(1761607681)));
  Val x280 = get(ctx, arg0, 163, 0);
  set(ctx, arg0, 164, (bitAnd(x260, Val(16)) * Val(1887436801)));
  Val x281 = get(ctx, arg0, 164, 0);
  set(ctx, arg0, 165, (bitAnd(x260, Val(32)) * Val(1950351361)));
  Val x282 = get(ctx, arg0, 165, 0);
  set(ctx, arg0, 166, (bitAnd(x260, Val(64)) * Val(1981808641)));
  Val x283 = get(ctx, arg0, 166, 0);
  set(ctx, arg0, 167, (bitAnd(x260, Val(128)) * Val(1997537281)));
  Val x284 = get(ctx, arg0, 167, 0);
  set(ctx, arg0, 168, (bitAnd(x260, Val(256)) * Val(2005401601)));
  Val x285 = get(ctx, arg0, 168, 0);
  set(ctx, arg0, 169, (bitAnd(x260, Val(512)) * Val(2009333761)));
  Val x286 = get(ctx, arg0, 169, 0);
  set(ctx, arg0, 170, (bitAnd(x260, Val(1024)) * Val(2011299841)));
  Val x287 = get(ctx, arg0, 170, 0);
  set(ctx, arg0, 171, (bitAnd(x260, Val(2048)) * Val(2012282881)));
  Val x288 = get(ctx, arg0, 171, 0);
  set(ctx, arg0, 172, (bitAnd(x260, Val(4096)) * Val(2012774401)));
  Val x289 = get(ctx, arg0, 172, 0);
  set(ctx, arg0, 173, (bitAnd(x260, Val(8192)) * Val(2013020161)));
  Val x290 = get(ctx, arg0, 173, 0);
  set(ctx, arg0, 174, (bitAnd(x260, Val(16384)) * Val(2013143041)));
  Val x291 = get(ctx, arg0, 174, 0);
  set(ctx, arg0, 175, (bitAnd(x260, Val(32768)) * Val(2013204481)));
  Val x292 = get(ctx, arg0, 175, 0);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:245)
  Val x293 = (x72 + get(ctx, arg0, 924, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x294 = (bitAnd(x293, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 961, bitAnd(x294, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 962, (bitAnd(x294, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 963, (bitAnd(x294, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x295 = ((get(ctx, arg0, 963, 0) * Val(4)) + (get(ctx, arg0, 962, 0) * Val(2)));
  Val x296 = (x295 + get(ctx, arg0, 961, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x297 = (x293 - (x296 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x298 = ((x80 + get(ctx, arg0, 925, 1)) + x296);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x299 = (bitAnd(x298, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 964, bitAnd(x299, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 965, (bitAnd(x299, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 966, (bitAnd(x299, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x300 = ((get(ctx, arg0, 966, 0) * Val(4)) + (get(ctx, arg0, 965, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x301 = (x298 - ((x300 + get(ctx, arg0, 964, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 496, bitAnd(x297, Val(1)));
  Val x302 = get(ctx, arg0, 496, 0);
  set(ctx, arg0, 497, (bitAnd(x297, Val(2)) * Val(1006632961)));
  Val x303 = get(ctx, arg0, 497, 0);
  set(ctx, arg0, 498, (bitAnd(x297, Val(4)) * Val(1509949441)));
  Val x304 = get(ctx, arg0, 498, 0);
  set(ctx, arg0, 499, (bitAnd(x297, Val(8)) * Val(1761607681)));
  Val x305 = get(ctx, arg0, 499, 0);
  set(ctx, arg0, 500, (bitAnd(x297, Val(16)) * Val(1887436801)));
  Val x306 = get(ctx, arg0, 500, 0);
  set(ctx, arg0, 501, (bitAnd(x297, Val(32)) * Val(1950351361)));
  Val x307 = get(ctx, arg0, 501, 0);
  set(ctx, arg0, 502, (bitAnd(x297, Val(64)) * Val(1981808641)));
  Val x308 = get(ctx, arg0, 502, 0);
  set(ctx, arg0, 503, (bitAnd(x297, Val(128)) * Val(1997537281)));
  Val x309 = get(ctx, arg0, 503, 0);
  set(ctx, arg0, 504, (bitAnd(x297, Val(256)) * Val(2005401601)));
  Val x310 = get(ctx, arg0, 504, 0);
  set(ctx, arg0, 505, (bitAnd(x297, Val(512)) * Val(2009333761)));
  Val x311 = get(ctx, arg0, 505, 0);
  set(ctx, arg0, 506, (bitAnd(x297, Val(1024)) * Val(2011299841)));
  Val x312 = get(ctx, arg0, 506, 0);
  set(ctx, arg0, 507, (bitAnd(x297, Val(2048)) * Val(2012282881)));
  Val x313 = get(ctx, arg0, 507, 0);
  set(ctx, arg0, 508, (bitAnd(x297, Val(4096)) * Val(2012774401)));
  Val x314 = get(ctx, arg0, 508, 0);
  set(ctx, arg0, 509, (bitAnd(x297, Val(8192)) * Val(2013020161)));
  Val x315 = get(ctx, arg0, 509, 0);
  set(ctx, arg0, 510, (bitAnd(x297, Val(16384)) * Val(2013143041)));
  Val x316 = get(ctx, arg0, 510, 0);
  set(ctx, arg0, 511, (bitAnd(x297, Val(32768)) * Val(2013204481)));
  Val x317 = get(ctx, arg0, 511, 0);
  set(ctx, arg0, 512, bitAnd(x301, Val(1)));
  Val x318 = get(ctx, arg0, 512, 0);
  set(ctx, arg0, 513, (bitAnd(x301, Val(2)) * Val(1006632961)));
  Val x319 = get(ctx, arg0, 513, 0);
  set(ctx, arg0, 514, (bitAnd(x301, Val(4)) * Val(1509949441)));
  Val x320 = get(ctx, arg0, 514, 0);
  set(ctx, arg0, 515, (bitAnd(x301, Val(8)) * Val(1761607681)));
  Val x321 = get(ctx, arg0, 515, 0);
  set(ctx, arg0, 516, (bitAnd(x301, Val(16)) * Val(1887436801)));
  Val x322 = get(ctx, arg0, 516, 0);
  set(ctx, arg0, 517, (bitAnd(x301, Val(32)) * Val(1950351361)));
  Val x323 = get(ctx, arg0, 517, 0);
  set(ctx, arg0, 518, (bitAnd(x301, Val(64)) * Val(1981808641)));
  Val x324 = get(ctx, arg0, 518, 0);
  set(ctx, arg0, 519, (bitAnd(x301, Val(128)) * Val(1997537281)));
  Val x325 = get(ctx, arg0, 519, 0);
  set(ctx, arg0, 520, (bitAnd(x301, Val(256)) * Val(2005401601)));
  Val x326 = get(ctx, arg0, 520, 0);
  set(ctx, arg0, 521, (bitAnd(x301, Val(512)) * Val(2009333761)));
  Val x327 = get(ctx, arg0, 521, 0);
  set(ctx, arg0, 522, (bitAnd(x301, Val(1024)) * Val(2011299841)));
  Val x328 = get(ctx, arg0, 522, 0);
  set(ctx, arg0, 523, (bitAnd(x301, Val(2048)) * Val(2012282881)));
  Val x329 = get(ctx, arg0, 523, 0);
  set(ctx, arg0, 524, (bitAnd(x301, Val(4096)) * Val(2012774401)));
  Val x330 = get(ctx, arg0, 524, 0);
  set(ctx, arg0, 525, (bitAnd(x301, Val(8192)) * Val(2013020161)));
  Val x331 = get(ctx, arg0, 525, 0);
  set(ctx, arg0, 526, (bitAnd(x301, Val(16384)) * Val(2013143041)));
  Val x332 = get(ctx, arg0, 526, 0);
  set(ctx, arg0, 527, (bitAnd(x301, Val(32768)) * Val(2013204481)));
  Val x333 = get(ctx, arg0, 527, 0);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x334 = (x88 + get(ctx, arg0, 926, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x335 = (bitAnd(x334, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 967, bitAnd(x335, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 968, (bitAnd(x335, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 969, (bitAnd(x335, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x336 = ((get(ctx, arg0, 969, 0) * Val(4)) + (get(ctx, arg0, 968, 0) * Val(2)));
  Val x337 = (x336 + get(ctx, arg0, 967, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x338 = (x334 - (x337 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x339 = ((x96 + get(ctx, arg0, 927, 1)) + x337);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x340 = (bitAnd(x339, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 970, bitAnd(x340, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 971, (bitAnd(x340, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 972, (bitAnd(x340, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x341 = ((get(ctx, arg0, 972, 0) * Val(4)) + (get(ctx, arg0, 971, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x342 = (x339 - ((x341 + get(ctx, arg0, 970, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 464, bitAnd(x338, Val(1)));
  Val x343 = get(ctx, arg0, 464, 0);
  set(ctx, arg0, 465, (bitAnd(x338, Val(2)) * Val(1006632961)));
  Val x344 = get(ctx, arg0, 465, 0);
  set(ctx, arg0, 466, (bitAnd(x338, Val(4)) * Val(1509949441)));
  Val x345 = get(ctx, arg0, 466, 0);
  set(ctx, arg0, 467, (bitAnd(x338, Val(8)) * Val(1761607681)));
  Val x346 = get(ctx, arg0, 467, 0);
  set(ctx, arg0, 468, (bitAnd(x338, Val(16)) * Val(1887436801)));
  Val x347 = get(ctx, arg0, 468, 0);
  set(ctx, arg0, 469, (bitAnd(x338, Val(32)) * Val(1950351361)));
  Val x348 = get(ctx, arg0, 469, 0);
  set(ctx, arg0, 470, (bitAnd(x338, Val(64)) * Val(1981808641)));
  Val x349 = get(ctx, arg0, 470, 0);
  set(ctx, arg0, 471, (bitAnd(x338, Val(128)) * Val(1997537281)));
  Val x350 = get(ctx, arg0, 471, 0);
  set(ctx, arg0, 472, (bitAnd(x338, Val(256)) * Val(2005401601)));
  Val x351 = get(ctx, arg0, 472, 0);
  set(ctx, arg0, 473, (bitAnd(x338, Val(512)) * Val(2009333761)));
  Val x352 = get(ctx, arg0, 473, 0);
  set(ctx, arg0, 474, (bitAnd(x338, Val(1024)) * Val(2011299841)));
  Val x353 = get(ctx, arg0, 474, 0);
  set(ctx, arg0, 475, (bitAnd(x338, Val(2048)) * Val(2012282881)));
  Val x354 = get(ctx, arg0, 475, 0);
  set(ctx, arg0, 476, (bitAnd(x338, Val(4096)) * Val(2012774401)));
  Val x355 = get(ctx, arg0, 476, 0);
  set(ctx, arg0, 477, (bitAnd(x338, Val(8192)) * Val(2013020161)));
  Val x356 = get(ctx, arg0, 477, 0);
  set(ctx, arg0, 478, (bitAnd(x338, Val(16384)) * Val(2013143041)));
  Val x357 = get(ctx, arg0, 478, 0);
  set(ctx, arg0, 479, (bitAnd(x338, Val(32768)) * Val(2013204481)));
  Val x358 = get(ctx, arg0, 479, 0);
  set(ctx, arg0, 480, bitAnd(x342, Val(1)));
  Val x359 = get(ctx, arg0, 480, 0);
  set(ctx, arg0, 481, (bitAnd(x342, Val(2)) * Val(1006632961)));
  Val x360 = get(ctx, arg0, 481, 0);
  set(ctx, arg0, 482, (bitAnd(x342, Val(4)) * Val(1509949441)));
  Val x361 = get(ctx, arg0, 482, 0);
  set(ctx, arg0, 483, (bitAnd(x342, Val(8)) * Val(1761607681)));
  Val x362 = get(ctx, arg0, 483, 0);
  set(ctx, arg0, 484, (bitAnd(x342, Val(16)) * Val(1887436801)));
  Val x363 = get(ctx, arg0, 484, 0);
  set(ctx, arg0, 485, (bitAnd(x342, Val(32)) * Val(1950351361)));
  Val x364 = get(ctx, arg0, 485, 0);
  set(ctx, arg0, 486, (bitAnd(x342, Val(64)) * Val(1981808641)));
  Val x365 = get(ctx, arg0, 486, 0);
  set(ctx, arg0, 487, (bitAnd(x342, Val(128)) * Val(1997537281)));
  Val x366 = get(ctx, arg0, 487, 0);
  set(ctx, arg0, 488, (bitAnd(x342, Val(256)) * Val(2005401601)));
  Val x367 = get(ctx, arg0, 488, 0);
  set(ctx, arg0, 489, (bitAnd(x342, Val(512)) * Val(2009333761)));
  Val x368 = get(ctx, arg0, 489, 0);
  set(ctx, arg0, 490, (bitAnd(x342, Val(1024)) * Val(2011299841)));
  Val x369 = get(ctx, arg0, 490, 0);
  set(ctx, arg0, 491, (bitAnd(x342, Val(2048)) * Val(2012282881)));
  Val x370 = get(ctx, arg0, 491, 0);
  set(ctx, arg0, 492, (bitAnd(x342, Val(4096)) * Val(2012774401)));
  Val x371 = get(ctx, arg0, 492, 0);
  set(ctx, arg0, 493, (bitAnd(x342, Val(8192)) * Val(2013020161)));
  Val x372 = get(ctx, arg0, 493, 0);
  set(ctx, arg0, 494, (bitAnd(x342, Val(16384)) * Val(2013143041)));
  Val x373 = get(ctx, arg0, 494, 0);
  set(ctx, arg0, 495, (bitAnd(x342, Val(32768)) * Val(2013204481)));
  Val x374 = get(ctx, arg0, 495, 0);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x375 = (x104 + get(ctx, arg0, 928, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x376 = (bitAnd(x375, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 973, bitAnd(x376, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 974, (bitAnd(x376, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 975, (bitAnd(x376, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x377 = ((get(ctx, arg0, 975, 0) * Val(4)) + (get(ctx, arg0, 974, 0) * Val(2)));
  Val x378 = (x377 + get(ctx, arg0, 973, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x379 = (x375 - (x378 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x380 = ((x112 + get(ctx, arg0, 929, 1)) + x378);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x381 = (bitAnd(x380, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 976, bitAnd(x381, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 977, (bitAnd(x381, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 978, (bitAnd(x381, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x382 = ((get(ctx, arg0, 978, 0) * Val(4)) + (get(ctx, arg0, 977, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x383 = (x380 - ((x382 + get(ctx, arg0, 976, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 432, bitAnd(x379, Val(1)));
  Val x384 = get(ctx, arg0, 432, 0);
  set(ctx, arg0, 433, (bitAnd(x379, Val(2)) * Val(1006632961)));
  Val x385 = get(ctx, arg0, 433, 0);
  set(ctx, arg0, 434, (bitAnd(x379, Val(4)) * Val(1509949441)));
  Val x386 = get(ctx, arg0, 434, 0);
  set(ctx, arg0, 435, (bitAnd(x379, Val(8)) * Val(1761607681)));
  Val x387 = get(ctx, arg0, 435, 0);
  set(ctx, arg0, 436, (bitAnd(x379, Val(16)) * Val(1887436801)));
  Val x388 = get(ctx, arg0, 436, 0);
  set(ctx, arg0, 437, (bitAnd(x379, Val(32)) * Val(1950351361)));
  Val x389 = get(ctx, arg0, 437, 0);
  set(ctx, arg0, 438, (bitAnd(x379, Val(64)) * Val(1981808641)));
  Val x390 = get(ctx, arg0, 438, 0);
  set(ctx, arg0, 439, (bitAnd(x379, Val(128)) * Val(1997537281)));
  Val x391 = get(ctx, arg0, 439, 0);
  set(ctx, arg0, 440, (bitAnd(x379, Val(256)) * Val(2005401601)));
  Val x392 = get(ctx, arg0, 440, 0);
  set(ctx, arg0, 441, (bitAnd(x379, Val(512)) * Val(2009333761)));
  Val x393 = get(ctx, arg0, 441, 0);
  set(ctx, arg0, 442, (bitAnd(x379, Val(1024)) * Val(2011299841)));
  Val x394 = get(ctx, arg0, 442, 0);
  set(ctx, arg0, 443, (bitAnd(x379, Val(2048)) * Val(2012282881)));
  Val x395 = get(ctx, arg0, 443, 0);
  set(ctx, arg0, 444, (bitAnd(x379, Val(4096)) * Val(2012774401)));
  Val x396 = get(ctx, arg0, 444, 0);
  set(ctx, arg0, 445, (bitAnd(x379, Val(8192)) * Val(2013020161)));
  Val x397 = get(ctx, arg0, 445, 0);
  set(ctx, arg0, 446, (bitAnd(x379, Val(16384)) * Val(2013143041)));
  Val x398 = get(ctx, arg0, 446, 0);
  set(ctx, arg0, 447, (bitAnd(x379, Val(32768)) * Val(2013204481)));
  Val x399 = get(ctx, arg0, 447, 0);
  set(ctx, arg0, 448, bitAnd(x383, Val(1)));
  Val x400 = get(ctx, arg0, 448, 0);
  set(ctx, arg0, 449, (bitAnd(x383, Val(2)) * Val(1006632961)));
  Val x401 = get(ctx, arg0, 449, 0);
  set(ctx, arg0, 450, (bitAnd(x383, Val(4)) * Val(1509949441)));
  Val x402 = get(ctx, arg0, 450, 0);
  set(ctx, arg0, 451, (bitAnd(x383, Val(8)) * Val(1761607681)));
  Val x403 = get(ctx, arg0, 451, 0);
  set(ctx, arg0, 452, (bitAnd(x383, Val(16)) * Val(1887436801)));
  Val x404 = get(ctx, arg0, 452, 0);
  set(ctx, arg0, 453, (bitAnd(x383, Val(32)) * Val(1950351361)));
  Val x405 = get(ctx, arg0, 453, 0);
  set(ctx, arg0, 454, (bitAnd(x383, Val(64)) * Val(1981808641)));
  Val x406 = get(ctx, arg0, 454, 0);
  set(ctx, arg0, 455, (bitAnd(x383, Val(128)) * Val(1997537281)));
  Val x407 = get(ctx, arg0, 455, 0);
  set(ctx, arg0, 456, (bitAnd(x383, Val(256)) * Val(2005401601)));
  Val x408 = get(ctx, arg0, 456, 0);
  set(ctx, arg0, 457, (bitAnd(x383, Val(512)) * Val(2009333761)));
  Val x409 = get(ctx, arg0, 457, 0);
  set(ctx, arg0, 458, (bitAnd(x383, Val(1024)) * Val(2011299841)));
  Val x410 = get(ctx, arg0, 458, 0);
  set(ctx, arg0, 459, (bitAnd(x383, Val(2048)) * Val(2012282881)));
  Val x411 = get(ctx, arg0, 459, 0);
  set(ctx, arg0, 460, (bitAnd(x383, Val(4096)) * Val(2012774401)));
  Val x412 = get(ctx, arg0, 460, 0);
  set(ctx, arg0, 461, (bitAnd(x383, Val(8192)) * Val(2013020161)));
  Val x413 = get(ctx, arg0, 461, 0);
  set(ctx, arg0, 462, (bitAnd(x383, Val(16384)) * Val(2013143041)));
  Val x414 = get(ctx, arg0, 462, 0);
  set(ctx, arg0, 463, (bitAnd(x383, Val(32768)) * Val(2013204481)));
  Val x415 = get(ctx, arg0, 463, 0);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x416 = (x120 + get(ctx, arg0, 930, 1));
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:42)
  Val x417 = (bitAnd(x416, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 979, bitAnd(x417, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 980, (bitAnd(x417, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 981, (bitAnd(x417, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x418 = ((get(ctx, arg0, 981, 0) * Val(4)) + (get(ctx, arg0, 980, 0) * Val(2)));
  Val x419 = (x418 + get(ctx, arg0, 979, 0));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x420 = (x416 - (x419 * Val(65536)));
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:43)
  Val x421 = ((x128 + get(ctx, arg0, 931, 1)) + x419);
  // Div(<preamble>:19)
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:34)
  Val x422 = (bitAnd(x421, Val(983040)) * Val(2013235201));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  set(ctx, arg0, 982, bitAnd(x422, Val(1)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:35)
  set(ctx, arg0, 983, (bitAnd(x422, Val(2)) * Val(1006632961)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:36)
  set(ctx, arg0, 984, (bitAnd(x422, Val(4)) * Val(1509949441)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:37)
  Val x423 = ((get(ctx, arg0, 984, 0) * Val(4)) + (get(ctx, arg0, 983, 0) * Val(2)));
  // CarryExtract(zirgen/circuit/keccak2/sha2.zir:38)
  Val x424 = (x421 - ((x423 + get(ctx, arg0, 982, 0)) * Val(65536)));
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  // UnpackReg(zirgen/circuit/keccak2/pack.zir:48)
  // CarryAndExpand(zirgen/circuit/keccak2/sha2.zir:44)
  set(ctx, arg0, 400, bitAnd(x420, Val(1)));
  Val x425 = get(ctx, arg0, 400, 0);
  set(ctx, arg0, 401, (bitAnd(x420, Val(2)) * Val(1006632961)));
  Val x426 = get(ctx, arg0, 401, 0);
  set(ctx, arg0, 402, (bitAnd(x420, Val(4)) * Val(1509949441)));
  Val x427 = get(ctx, arg0, 402, 0);
  set(ctx, arg0, 403, (bitAnd(x420, Val(8)) * Val(1761607681)));
  Val x428 = get(ctx, arg0, 403, 0);
  set(ctx, arg0, 404, (bitAnd(x420, Val(16)) * Val(1887436801)));
  Val x429 = get(ctx, arg0, 404, 0);
  set(ctx, arg0, 405, (bitAnd(x420, Val(32)) * Val(1950351361)));
  Val x430 = get(ctx, arg0, 405, 0);
  set(ctx, arg0, 406, (bitAnd(x420, Val(64)) * Val(1981808641)));
  Val x431 = get(ctx, arg0, 406, 0);
  set(ctx, arg0, 407, (bitAnd(x420, Val(128)) * Val(1997537281)));
  Val x432 = get(ctx, arg0, 407, 0);
  set(ctx, arg0, 408, (bitAnd(x420, Val(256)) * Val(2005401601)));
  Val x433 = get(ctx, arg0, 408, 0);
  set(ctx, arg0, 409, (bitAnd(x420, Val(512)) * Val(2009333761)));
  Val x434 = get(ctx, arg0, 409, 0);
  set(ctx, arg0, 410, (bitAnd(x420, Val(1024)) * Val(2011299841)));
  Val x435 = get(ctx, arg0, 410, 0);
  set(ctx, arg0, 411, (bitAnd(x420, Val(2048)) * Val(2012282881)));
  Val x436 = get(ctx, arg0, 411, 0);
  set(ctx, arg0, 412, (bitAnd(x420, Val(4096)) * Val(2012774401)));
  Val x437 = get(ctx, arg0, 412, 0);
  set(ctx, arg0, 413, (bitAnd(x420, Val(8192)) * Val(2013020161)));
  Val x438 = get(ctx, arg0, 413, 0);
  set(ctx, arg0, 414, (bitAnd(x420, Val(16384)) * Val(2013143041)));
  Val x439 = get(ctx, arg0, 414, 0);
  set(ctx, arg0, 415, (bitAnd(x420, Val(32768)) * Val(2013204481)));
  Val x440 = get(ctx, arg0, 415, 0);
  set(ctx, arg0, 416, bitAnd(x424, Val(1)));
  Val x441 = get(ctx, arg0, 416, 0);
  set(ctx, arg0, 417, (bitAnd(x424, Val(2)) * Val(1006632961)));
  Val x442 = get(ctx, arg0, 417, 0);
  set(ctx, arg0, 418, (bitAnd(x424, Val(4)) * Val(1509949441)));
  Val x443 = get(ctx, arg0, 418, 0);
  set(ctx, arg0, 419, (bitAnd(x424, Val(8)) * Val(1761607681)));
  Val x444 = get(ctx, arg0, 419, 0);
  set(ctx, arg0, 420, (bitAnd(x424, Val(16)) * Val(1887436801)));
  Val x445 = get(ctx, arg0, 420, 0);
  set(ctx, arg0, 421, (bitAnd(x424, Val(32)) * Val(1950351361)));
  Val x446 = get(ctx, arg0, 421, 0);
  set(ctx, arg0, 422, (bitAnd(x424, Val(64)) * Val(1981808641)));
  Val x447 = get(ctx, arg0, 422, 0);
  set(ctx, arg0, 423, (bitAnd(x424, Val(128)) * Val(1997537281)));
  Val x448 = get(ctx, arg0, 423, 0);
  set(ctx, arg0, 424, (bitAnd(x424, Val(256)) * Val(2005401601)));
  Val x449 = get(ctx, arg0, 424, 0);
  set(ctx, arg0, 425, (bitAnd(x424, Val(512)) * Val(2009333761)));
  Val x450 = get(ctx, arg0, 425, 0);
  set(ctx, arg0, 426, (bitAnd(x424, Val(1024)) * Val(2011299841)));
  Val x451 = get(ctx, arg0, 426, 0);
  set(ctx, arg0, 427, (bitAnd(x424, Val(2048)) * Val(2012282881)));
  Val x452 = get(ctx, arg0, 427, 0);
  set(ctx, arg0, 428, (bitAnd(x424, Val(4096)) * Val(2012774401)));
  Val x453 = get(ctx, arg0, 428, 0);
  set(ctx, arg0, 429, (bitAnd(x424, Val(8192)) * Val(2013020161)));
  Val x454 = get(ctx, arg0, 429, 0);
  set(ctx, arg0, 430, (bitAnd(x424, Val(16384)) * Val(2013143041)));
  Val x455 = get(ctx, arg0, 430, 0);
  set(ctx, arg0, 431, (bitAnd(x424, Val(32768)) * Val(2013204481)));
  Val x456 = get(ctx, arg0, 431, 0);
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:55)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:257)
  Val x457 = (((x138 + (x139 * Val(2))) + (x140 * Val(4))) + (x141 * Val(8)));
  Val x458 = (((x457 + (x142 * Val(16))) + (x143 * Val(32))) + (x144 * Val(64)));
  Val x459 = (((x458 + (x145 * Val(128))) + (x146 * Val(256))) + (x147 * Val(512)));
  Val x460 = (((x459 + (x148 * Val(1024))) + (x149 * Val(2048))) + (x150 * Val(4096)));
  Val x461 = (((x460 + (x151 * Val(8192))) + (x152 * Val(16384))) + (x153 * Val(32768)));
  Val x462 = (((x154 + (x155 * Val(2))) + (x156 * Val(4))) + (x157 * Val(8)));
  Val x463 = (((x462 + (x158 * Val(16))) + (x159 * Val(32))) + (x160 * Val(64)));
  Val x464 = (((x463 + (x161 * Val(128))) + (x162 * Val(256))) + (x163 * Val(512)));
  Val x465 = (((x464 + (x164 * Val(1024))) + (x165 * Val(2048))) + (x166 * Val(4096)));
  Val x466 = (((x465 + (x167 * Val(8192))) + (x168 * Val(16384))) + (x169 * Val(32768)));
  Val x467 = (((x179 + (x180 * Val(2))) + (x181 * Val(4))) + (x182 * Val(8)));
  Val x468 = (((x467 + (x183 * Val(16))) + (x184 * Val(32))) + (x185 * Val(64)));
  Val x469 = (((x468 + (x186 * Val(128))) + (x187 * Val(256))) + (x188 * Val(512)));
  Val x470 = (((x469 + (x189 * Val(1024))) + (x190 * Val(2048))) + (x191 * Val(4096)));
  Val x471 = (((x470 + (x192 * Val(8192))) + (x193 * Val(16384))) + (x194 * Val(32768)));
  Val x472 = (((x195 + (x196 * Val(2))) + (x197 * Val(4))) + (x198 * Val(8)));
  Val x473 = (((x472 + (x199 * Val(16))) + (x200 * Val(32))) + (x201 * Val(64)));
  Val x474 = (((x473 + (x202 * Val(128))) + (x203 * Val(256))) + (x204 * Val(512)));
  Val x475 = (((x474 + (x205 * Val(1024))) + (x206 * Val(2048))) + (x207 * Val(4096)));
  Val x476 = (((x475 + (x208 * Val(8192))) + (x209 * Val(16384))) + (x210 * Val(32768)));
  Val x477 = (((x220 + (x221 * Val(2))) + (x222 * Val(4))) + (x223 * Val(8)));
  Val x478 = (((x477 + (x224 * Val(16))) + (x225 * Val(32))) + (x226 * Val(64)));
  Val x479 = (((x478 + (x227 * Val(128))) + (x228 * Val(256))) + (x229 * Val(512)));
  Val x480 = (((x479 + (x230 * Val(1024))) + (x231 * Val(2048))) + (x232 * Val(4096)));
  Val x481 = (((x480 + (x233 * Val(8192))) + (x234 * Val(16384))) + (x235 * Val(32768)));
  Val x482 = (((x236 + (x237 * Val(2))) + (x238 * Val(4))) + (x239 * Val(8)));
  Val x483 = (((x482 + (x240 * Val(16))) + (x241 * Val(32))) + (x242 * Val(64)));
  Val x484 = (((x483 + (x243 * Val(128))) + (x244 * Val(256))) + (x245 * Val(512)));
  Val x485 = (((x484 + (x246 * Val(1024))) + (x247 * Val(2048))) + (x248 * Val(4096)));
  Val x486 = (((x485 + (x249 * Val(8192))) + (x250 * Val(16384))) + (x251 * Val(32768)));
  Val x487 = (((x261 + (x262 * Val(2))) + (x263 * Val(4))) + (x264 * Val(8)));
  Val x488 = (((x487 + (x265 * Val(16))) + (x266 * Val(32))) + (x267 * Val(64)));
  Val x489 = (((x488 + (x268 * Val(128))) + (x269 * Val(256))) + (x270 * Val(512)));
  Val x490 = (((x489 + (x271 * Val(1024))) + (x272 * Val(2048))) + (x273 * Val(4096)));
  Val x491 = (((x490 + (x274 * Val(8192))) + (x275 * Val(16384))) + (x276 * Val(32768)));
  Val x492 = (((x277 + (x278 * Val(2))) + (x279 * Val(4))) + (x280 * Val(8)));
  Val x493 = (((x492 + (x281 * Val(16))) + (x282 * Val(32))) + (x283 * Val(64)));
  Val x494 = (((x493 + (x284 * Val(128))) + (x285 * Val(256))) + (x286 * Val(512)));
  Val x495 = (((x494 + (x287 * Val(1024))) + (x288 * Val(2048))) + (x289 * Val(4096)));
  Val x496 = (((x495 + (x290 * Val(8192))) + (x291 * Val(16384))) + (x292 * Val(32768)));
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:259)
  Val x497 = (((x302 + (x303 * Val(2))) + (x304 * Val(4))) + (x305 * Val(8)));
  Val x498 = (((x497 + (x306 * Val(16))) + (x307 * Val(32))) + (x308 * Val(64)));
  Val x499 = (((x498 + (x309 * Val(128))) + (x310 * Val(256))) + (x311 * Val(512)));
  Val x500 = (((x499 + (x312 * Val(1024))) + (x313 * Val(2048))) + (x314 * Val(4096)));
  Val x501 = (((x500 + (x315 * Val(8192))) + (x316 * Val(16384))) + (x317 * Val(32768)));
  Val x502 = (((x318 + (x319 * Val(2))) + (x320 * Val(4))) + (x321 * Val(8)));
  Val x503 = (((x502 + (x322 * Val(16))) + (x323 * Val(32))) + (x324 * Val(64)));
  Val x504 = (((x503 + (x325 * Val(128))) + (x326 * Val(256))) + (x327 * Val(512)));
  Val x505 = (((x504 + (x328 * Val(1024))) + (x329 * Val(2048))) + (x330 * Val(4096)));
  Val x506 = (((x505 + (x331 * Val(8192))) + (x332 * Val(16384))) + (x333 * Val(32768)));
  Val x507 = (((x343 + (x344 * Val(2))) + (x345 * Val(4))) + (x346 * Val(8)));
  Val x508 = (((x507 + (x347 * Val(16))) + (x348 * Val(32))) + (x349 * Val(64)));
  Val x509 = (((x508 + (x350 * Val(128))) + (x351 * Val(256))) + (x352 * Val(512)));
  Val x510 = (((x509 + (x353 * Val(1024))) + (x354 * Val(2048))) + (x355 * Val(4096)));
  Val x511 = (((x510 + (x356 * Val(8192))) + (x357 * Val(16384))) + (x358 * Val(32768)));
  Val x512 = (((x359 + (x360 * Val(2))) + (x361 * Val(4))) + (x362 * Val(8)));
  Val x513 = (((x512 + (x363 * Val(16))) + (x364 * Val(32))) + (x365 * Val(64)));
  Val x514 = (((x513 + (x366 * Val(128))) + (x367 * Val(256))) + (x368 * Val(512)));
  Val x515 = (((x514 + (x369 * Val(1024))) + (x370 * Val(2048))) + (x371 * Val(4096)));
  Val x516 = (((x515 + (x372 * Val(8192))) + (x373 * Val(16384))) + (x374 * Val(32768)));
  Val x517 = (((x384 + (x385 * Val(2))) + (x386 * Val(4))) + (x387 * Val(8)));
  Val x518 = (((x517 + (x388 * Val(16))) + (x389 * Val(32))) + (x390 * Val(64)));
  Val x519 = (((x518 + (x391 * Val(128))) + (x392 * Val(256))) + (x393 * Val(512)));
  Val x520 = (((x519 + (x394 * Val(1024))) + (x395 * Val(2048))) + (x396 * Val(4096)));
  Val x521 = (((x520 + (x397 * Val(8192))) + (x398 * Val(16384))) + (x399 * Val(32768)));
  Val x522 = (((x400 + (x401 * Val(2))) + (x402 * Val(4))) + (x403 * Val(8)));
  Val x523 = (((x522 + (x404 * Val(16))) + (x405 * Val(32))) + (x406 * Val(64)));
  Val x524 = (((x523 + (x407 * Val(128))) + (x408 * Val(256))) + (x409 * Val(512)));
  Val x525 = (((x524 + (x410 * Val(1024))) + (x411 * Val(2048))) + (x412 * Val(4096)));
  Val x526 = (((x525 + (x413 * Val(8192))) + (x414 * Val(16384))) + (x415 * Val(32768)));
  Val x527 = (((x425 + (x426 * Val(2))) + (x427 * Val(4))) + (x428 * Val(8)));
  Val x528 = (((x527 + (x429 * Val(16))) + (x430 * Val(32))) + (x431 * Val(64)));
  Val x529 = (((x528 + (x432 * Val(128))) + (x433 * Val(256))) + (x434 * Val(512)));
  Val x530 = (((x529 + (x435 * Val(1024))) + (x436 * Val(2048))) + (x437 * Val(4096)));
  Val x531 = (((x530 + (x438 * Val(8192))) + (x439 * Val(16384))) + (x440 * Val(32768)));
  Val x532 = (((x441 + (x442 * Val(2))) + (x443 * Val(4))) + (x444 * Val(8)));
  Val x533 = (((x532 + (x445 * Val(16))) + (x446 * Val(32))) + (x447 * Val(64)));
  Val x534 = (((x533 + (x448 * Val(128))) + (x449 * Val(256))) + (x450 * Val(512)));
  Val x535 = (((x534 + (x451 * Val(1024))) + (x452 * Val(2048))) + (x453 * Val(4096)));
  Val x536 = (((x535 + (x454 * Val(8192))) + (x455 * Val(16384))) + (x456 * Val(32768)));
  // Reg(<preamble>:4)
  // TopState(zirgen/circuit/keccak2/top.zir:36)
  // ShaNextBlockCycle(zirgen/circuit/keccak2/top.zir:262)
  set(ctx, arg0, 16, Val(0));
  set(ctx, arg0, 17, Val(0));
  set(ctx, arg0, 18, Val(0));
  set(ctx, arg0, 19, Val(0));
  set(ctx, arg0, 20, Val(0));
  set(ctx, arg0, 21, Val(0));
  set(ctx, arg0, 22, Val(0));
  set(ctx, arg0, 23, Val(0));
  set(ctx, arg0, 24, Val(0));
  set(ctx, arg0, 25, Val(0));
  set(ctx, arg0, 26, Val(0));
  set(ctx, arg0, 27, Val(0));
  set(ctx, arg0, 28, Val(0));
  set(ctx, arg0, 29, Val(0));
  set(ctx, arg0, 30, Val(0));
  set(ctx, arg0, 31, Val(0));
  set(ctx, arg0, 32, Val(0));
  set(ctx, arg0, 33, Val(0));
  set(ctx, arg0, 34, Val(0));
  set(ctx, arg0, 35, Val(0));
  set(ctx, arg0, 36, Val(0));
  set(ctx, arg0, 37, Val(0));
  set(ctx, arg0, 38, Val(0));
  set(ctx, arg0, 39, Val(0));
  set(ctx, arg0, 40, Val(0));
  set(ctx, arg0, 41, Val(0));
  set(ctx, arg0, 42, Val(0));
  set(ctx, arg0, 43, Val(0));
  set(ctx, arg0, 44, Val(0));
  set(ctx, arg0, 45, Val(0));
  set(ctx, arg0, 46, Val(0));
  set(ctx, arg0, 47, Val(0));
  set(ctx, arg0, 48, Val(0));
  set(ctx, arg0, 49, Val(0));
  set(ctx, arg0, 50, Val(0));
  set(ctx, arg0, 51, Val(0));
  set(ctx, arg0, 52, Val(0));
  set(ctx, arg0, 53, Val(0));
  set(ctx, arg0, 54, Val(0));
  set(ctx, arg0, 55, Val(0));
  set(ctx, arg0, 56, Val(0));
  set(ctx, arg0, 57, Val(0));
  set(ctx, arg0, 58, Val(0));
  set(ctx, arg0, 59, Val(0));
  set(ctx, arg0, 60, Val(0));
  set(ctx, arg0, 61, Val(0));
  set(ctx, arg0, 62, Val(0));
  set(ctx, arg0, 63, Val(0));
  set(ctx, arg0, 64, Val(0));
  set(ctx, arg0, 65, Val(0));
  set(ctx, arg0, 66, Val(0));
  set(ctx, arg0, 67, Val(0));
  set(ctx, arg0, 68, Val(0));
  set(ctx, arg0, 69, Val(0));
  set(ctx, arg0, 70, Val(0));
  set(ctx, arg0, 71, Val(0));
  set(ctx, arg0, 72, Val(0));
  set(ctx, arg0, 73, Val(0));
  set(ctx, arg0, 74, Val(0));
  set(ctx, arg0, 75, Val(0));
  set(ctx, arg0, 76, Val(0));
  set(ctx, arg0, 77, Val(0));
  set(ctx, arg0, 78, Val(0));
  set(ctx, arg0, 79, Val(0));
  set(ctx, arg0, 80, Val(0));
  set(ctx, arg0, 81, Val(0));
  set(ctx, arg0, 82, Val(0));
  set(ctx, arg0, 83, Val(0));
  set(ctx, arg0, 84, Val(0));
  set(ctx, arg0, 85, Val(0));
  set(ctx, arg0, 86, Val(0));
  set(ctx, arg0, 87, Val(0));
  set(ctx, arg0, 88, Val(0));
  set(ctx, arg0, 89, Val(0));
  set(ctx, arg0, 90, Val(0));
  set(ctx, arg0, 91, Val(0));
  set(ctx, arg0, 92, Val(0));
  set(ctx, arg0, 93, Val(0));
  set(ctx, arg0, 94, Val(0));
  set(ctx, arg0, 95, Val(0));
  set(ctx, arg0, 96, Val(0));
  set(ctx, arg0, 97, Val(0));
  set(ctx, arg0, 98, Val(0));
  set(ctx, arg0, 99, Val(0));
  set(ctx, arg0, 100, Val(0));
  set(ctx, arg0, 101, Val(0));
  set(ctx, arg0, 102, Val(0));
  set(ctx, arg0, 103, Val(0));
  set(ctx, arg0, 104, Val(0));
  set(ctx, arg0, 105, Val(0));
  set(ctx, arg0, 106, Val(0));
  set(ctx, arg0, 107, Val(0));
  set(ctx, arg0, 108, Val(0));
  set(ctx, arg0, 109, Val(0));
  set(ctx, arg0, 110, Val(0));
  set(ctx, arg0, 111, Val(0));
  set(ctx, arg0, 112, Val(0));
  set(ctx, arg0, 113, Val(0));
  set(ctx, arg0, 114, Val(0));
  set(ctx, arg0, 115, Val(0));
  set(ctx, arg0, 116, Val(0));
  set(ctx, arg0, 117, Val(0));
  set(ctx, arg0, 118, Val(0));
  set(ctx, arg0, 119, Val(0));
  set(ctx, arg0, 120, Val(0));
  set(ctx, arg0, 121, Val(0));
  set(ctx, arg0, 122, Val(0));
  set(ctx, arg0, 123, Val(0));
  set(ctx, arg0, 124, Val(0));
  set(ctx, arg0, 125, Val(0));
  set(ctx, arg0, 126, Val(0));
  set(ctx, arg0, 127, Val(0));
  set(ctx, arg0, 128, Val(0));
  set(ctx, arg0, 129, Val(0));
  set(ctx, arg0, 130, Val(0));
  set(ctx, arg0, 131, Val(0));
  set(ctx, arg0, 132, Val(0));
  set(ctx, arg0, 133, Val(0));
  set(ctx, arg0, 134, Val(0));
  set(ctx, arg0, 135, Val(0));
  set(ctx, arg0, 136, Val(0));
  set(ctx, arg0, 137, Val(0));
  set(ctx, arg0, 138, Val(0));
  set(ctx, arg0, 139, Val(0));
  set(ctx, arg0, 140, Val(0));
  set(ctx, arg0, 141, Val(0));
  set(ctx, arg0, 142, Val(0));
  set(ctx, arg0, 143, Val(0));
  set(ctx, arg0, 144, x261);
  set(ctx, arg0, 145, x262);
  set(ctx, arg0, 146, x263);
  set(ctx, arg0, 147, x264);
  set(ctx, arg0, 148, x265);
  set(ctx, arg0, 149, x266);
  set(ctx, arg0, 150, x267);
  set(ctx, arg0, 151, x268);
  set(ctx, arg0, 152, x269);
  set(ctx, arg0, 153, x270);
  set(ctx, arg0, 154, x271);
  set(ctx, arg0, 155, x272);
  set(ctx, arg0, 156, x273);
  set(ctx, arg0, 157, x274);
  set(ctx, arg0, 158, x275);
  set(ctx, arg0, 159, x276);
  set(ctx, arg0, 160, x277);
  set(ctx, arg0, 161, x278);
  set(ctx, arg0, 162, x279);
  set(ctx, arg0, 163, x280);
  set(ctx, arg0, 164, x281);
  set(ctx, arg0, 165, x282);
  set(ctx, arg0, 166, x283);
  set(ctx, arg0, 167, x284);
  set(ctx, arg0, 168, x285);
  set(ctx, arg0, 169, x286);
  set(ctx, arg0, 170, x287);
  set(ctx, arg0, 171, x288);
  set(ctx, arg0, 172, x289);
  set(ctx, arg0, 173, x290);
  set(ctx, arg0, 174, x291);
  set(ctx, arg0, 175, x292);
  set(ctx, arg0, 176, x220);
  set(ctx, arg0, 177, x221);
  set(ctx, arg0, 178, x222);
  set(ctx, arg0, 179, x223);
  set(ctx, arg0, 180, x224);
  set(ctx, arg0, 181, x225);
  set(ctx, arg0, 182, x226);
  set(ctx, arg0, 183, x227);
  set(ctx, arg0, 184, x228);
  set(ctx, arg0, 185, x229);
  set(ctx, arg0, 186, x230);
  set(ctx, arg0, 187, x231);
  set(ctx, arg0, 188, x232);
  set(ctx, arg0, 189, x233);
  set(ctx, arg0, 190, x234);
  set(ctx, arg0, 191, x235);
  set(ctx, arg0, 192, x236);
  set(ctx, arg0, 193, x237);
  set(ctx, arg0, 194, x238);
  set(ctx, arg0, 195, x239);
  set(ctx, arg0, 196, x240);
  set(ctx, arg0, 197, x241);
  set(ctx, arg0, 198, x242);
  set(ctx, arg0, 199, x243);
  set(ctx, arg0, 200, x244);
  set(ctx, arg0, 201, x245);
  set(ctx, arg0, 202, x246);
  set(ctx, arg0, 203, x247);
  set(ctx, arg0, 204, x248);
  set(ctx, arg0, 205, x249);
  set(ctx, arg0, 206, x250);
  set(ctx, arg0, 207, x251);
  set(ctx, arg0, 208, x179);
  set(ctx, arg0, 209, x180);
  set(ctx, arg0, 210, x181);
  set(ctx, arg0, 211, x182);
  set(ctx, arg0, 212, x183);
  set(ctx, arg0, 213, x184);
  set(ctx, arg0, 214, x185);
  set(ctx, arg0, 215, x186);
  set(ctx, arg0, 216, x187);
  set(ctx, arg0, 217, x188);
  set(ctx, arg0, 218, x189);
  set(ctx, arg0, 219, x190);
  set(ctx, arg0, 220, x191);
  set(ctx, arg0, 221, x192);
  set(ctx, arg0, 222, x193);
  set(ctx, arg0, 223, x194);
  set(ctx, arg0, 224, x195);
  set(ctx, arg0, 225, x196);
  set(ctx, arg0, 226, x197);
  set(ctx, arg0, 227, x198);
  set(ctx, arg0, 228, x199);
  set(ctx, arg0, 229, x200);
  set(ctx, arg0, 230, x201);
  set(ctx, arg0, 231, x202);
  set(ctx, arg0, 232, x203);
  set(ctx, arg0, 233, x204);
  set(ctx, arg0, 234, x205);
  set(ctx, arg0, 235, x206);
  set(ctx, arg0, 236, x207);
  set(ctx, arg0, 237, x208);
  set(ctx, arg0, 238, x209);
  set(ctx, arg0, 239, x210);
  set(ctx, arg0, 240, x138);
  set(ctx, arg0, 241, x139);
  set(ctx, arg0, 242, x140);
  set(ctx, arg0, 243, x141);
  set(ctx, arg0, 244, x142);
  set(ctx, arg0, 245, x143);
  set(ctx, arg0, 246, x144);
  set(ctx, arg0, 247, x145);
  set(ctx, arg0, 248, x146);
  set(ctx, arg0, 249, x147);
  set(ctx, arg0, 250, x148);
  set(ctx, arg0, 251, x149);
  set(ctx, arg0, 252, x150);
  set(ctx, arg0, 253, x151);
  set(ctx, arg0, 254, x152);
  set(ctx, arg0, 255, x153);
  set(ctx, arg0, 256, x154);
  set(ctx, arg0, 257, x155);
  set(ctx, arg0, 258, x156);
  set(ctx, arg0, 259, x157);
  set(ctx, arg0, 260, x158);
  set(ctx, arg0, 261, x159);
  set(ctx, arg0, 262, x160);
  set(ctx, arg0, 263, x161);
  set(ctx, arg0, 264, x162);
  set(ctx, arg0, 265, x163);
  set(ctx, arg0, 266, x164);
  set(ctx, arg0, 267, x165);
  set(ctx, arg0, 268, x166);
  set(ctx, arg0, 269, x167);
  set(ctx, arg0, 270, x168);
  set(ctx, arg0, 271, x169);
  set(ctx, arg0, 272, Val(0));
  set(ctx, arg0, 273, Val(0));
  set(ctx, arg0, 274, Val(0));
  set(ctx, arg0, 275, Val(0));
  set(ctx, arg0, 276, Val(0));
  set(ctx, arg0, 277, Val(0));
  set(ctx, arg0, 278, Val(0));
  set(ctx, arg0, 279, Val(0));
  set(ctx, arg0, 280, Val(0));
  set(ctx, arg0, 281, Val(0));
  set(ctx, arg0, 282, Val(0));
  set(ctx, arg0, 283, Val(0));
  set(ctx, arg0, 284, Val(0));
  set(ctx, arg0, 285, Val(0));
  set(ctx, arg0, 286, Val(0));
  set(ctx, arg0, 287, Val(0));
  set(ctx, arg0, 288, Val(0));
  set(ctx, arg0, 289, Val(0));
  set(ctx, arg0, 290, Val(0));
  set(ctx, arg0, 291, Val(0));
  set(ctx, arg0, 292, Val(0));
  set(ctx, arg0, 293, Val(0));
  set(ctx, arg0, 294, Val(0));
  set(ctx, arg0, 295, Val(0));
  set(ctx, arg0, 296, Val(0));
  set(ctx, arg0, 297, Val(0));
  set(ctx, arg0, 298, Val(0));
  set(ctx, arg0, 299, Val(0));
  set(ctx, arg0, 300, Val(0));
  set(ctx, arg0, 301, Val(0));
  set(ctx, arg0, 302, Val(0));
  set(ctx, arg0, 303, Val(0));
  set(ctx, arg0, 304, Val(0));
  set(ctx, arg0, 305, Val(0));
  set(ctx, arg0, 306, Val(0));
  set(ctx, arg0, 307, Val(0));
  set(ctx, arg0, 308, Val(0));
  set(ctx, arg0, 309, Val(0));
  set(ctx, arg0, 310, Val(0));
  set(ctx, arg0, 311, Val(0));
  set(ctx, arg0, 312, Val(0));
  set(ctx, arg0, 313, Val(0));
  set(ctx, arg0, 314, Val(0));
  set(ctx, arg0, 315, Val(0));
  set(ctx, arg0, 316, Val(0));
  set(ctx, arg0, 317, Val(0));
  set(ctx, arg0, 318, Val(0));
  set(ctx, arg0, 319, Val(0));
  set(ctx, arg0, 320, Val(0));
  set(ctx, arg0, 321, Val(0));
  set(ctx, arg0, 322, Val(0));
  set(ctx, arg0, 323, Val(0));
  set(ctx, arg0, 324, Val(0));
  set(ctx, arg0, 325, Val(0));
  set(ctx, arg0, 326, Val(0));
  set(ctx, arg0, 327, Val(0));
  set(ctx, arg0, 328, Val(0));
  set(ctx, arg0, 329, Val(0));
  set(ctx, arg0, 330, Val(0));
  set(ctx, arg0, 331, Val(0));
  set(ctx, arg0, 332, Val(0));
  set(ctx, arg0, 333, Val(0));
  set(ctx, arg0, 334, Val(0));
  set(ctx, arg0, 335, Val(0));
  set(ctx, arg0, 336, Val(0));
  set(ctx, arg0, 337, Val(0));
  set(ctx, arg0, 338, Val(0));
  set(ctx, arg0, 339, Val(0));
  set(ctx, arg0, 340, Val(0));
  set(ctx, arg0, 341, Val(0));
  set(ctx, arg0, 342, Val(0));
  set(ctx, arg0, 343, Val(0));
  set(ctx, arg0, 344, Val(0));
  set(ctx, arg0, 345, Val(0));
  set(ctx, arg0, 346, Val(0));
  set(ctx, arg0, 347, Val(0));
  set(ctx, arg0, 348, Val(0));
  set(ctx, arg0, 349, Val(0));
  set(ctx, arg0, 350, Val(0));
  set(ctx, arg0, 351, Val(0));
  set(ctx, arg0, 352, Val(0));
  set(ctx, arg0, 353, Val(0));
  set(ctx, arg0, 354, Val(0));
  set(ctx, arg0, 355, Val(0));
  set(ctx, arg0, 356, Val(0));
  set(ctx, arg0, 357, Val(0));
  set(ctx, arg0, 358, Val(0));
  set(ctx, arg0, 359, Val(0));
  set(ctx, arg0, 360, Val(0));
  set(ctx, arg0, 361, Val(0));
  set(ctx, arg0, 362, Val(0));
  set(ctx, arg0, 363, Val(0));
  set(ctx, arg0, 364, Val(0));
  set(ctx, arg0, 365, Val(0));
  set(ctx, arg0, 366, Val(0));
  set(ctx, arg0, 367, Val(0));
  set(ctx, arg0, 368, Val(0));
  set(ctx, arg0, 369, Val(0));
  set(ctx, arg0, 370, Val(0));
  set(ctx, arg0, 371, Val(0));
  set(ctx, arg0, 372, Val(0));
  set(ctx, arg0, 373, Val(0));
  set(ctx, arg0, 374, Val(0));
  set(ctx, arg0, 375, Val(0));
  set(ctx, arg0, 376, Val(0));
  set(ctx, arg0, 377, Val(0));
  set(ctx, arg0, 378, Val(0));
  set(ctx, arg0, 379, Val(0));
  set(ctx, arg0, 380, Val(0));
  set(ctx, arg0, 381, Val(0));
  set(ctx, arg0, 382, Val(0));
  set(ctx, arg0, 383, Val(0));
  set(ctx, arg0, 384, Val(0));
  set(ctx, arg0, 385, Val(0));
  set(ctx, arg0, 386, Val(0));
  set(ctx, arg0, 387, Val(0));
  set(ctx, arg0, 388, Val(0));
  set(ctx, arg0, 389, Val(0));
  set(ctx, arg0, 390, Val(0));
  set(ctx, arg0, 391, Val(0));
  set(ctx, arg0, 392, Val(0));
  set(ctx, arg0, 393, Val(0));
  set(ctx, arg0, 394, Val(0));
  set(ctx, arg0, 395, Val(0));
  set(ctx, arg0, 396, Val(0));
  set(ctx, arg0, 397, Val(0));
  set(ctx, arg0, 398, Val(0));
  set(ctx, arg0, 399, Val(0));
  set(ctx, arg0, 400, x425);
  set(ctx, arg0, 401, x426);
  set(ctx, arg0, 402, x427);
  set(ctx, arg0, 403, x428);
  set(ctx, arg0, 404, x429);
  set(ctx, arg0, 405, x430);
  set(ctx, arg0, 406, x431);
  set(ctx, arg0, 407, x432);
  set(ctx, arg0, 408, x433);
  set(ctx, arg0, 409, x434);
  set(ctx, arg0, 410, x435);
  set(ctx, arg0, 411, x436);
  set(ctx, arg0, 412, x437);
  set(ctx, arg0, 413, x438);
  set(ctx, arg0, 414, x439);
  set(ctx, arg0, 415, x440);
  set(ctx, arg0, 416, x441);
  set(ctx, arg0, 417, x442);
  set(ctx, arg0, 418, x443);
  set(ctx, arg0, 419, x444);
  set(ctx, arg0, 420, x445);
  set(ctx, arg0, 421, x446);
  set(ctx, arg0, 422, x447);
  set(ctx, arg0, 423, x448);
  set(ctx, arg0, 424, x449);
  set(ctx, arg0, 425, x450);
  set(ctx, arg0, 426, x451);
  set(ctx, arg0, 427, x452);
  set(ctx, arg0, 428, x453);
  set(ctx, arg0, 429, x454);
  set(ctx, arg0, 430, x455);
  set(ctx, arg0, 431, x456);
  set(ctx, arg0, 432, x384);
  set(ctx, arg0, 433, x385);
  set(ctx, arg0, 434, x386);
  set(ctx, arg0, 435, x387);
  set(ctx, arg0, 436, x388);
  set(ctx, arg0, 437, x389);
  set(ctx, arg0, 438, x390);
  set(ctx, arg0, 439, x391);
  set(ctx, arg0, 440, x392);
  set(ctx, arg0, 441, x393);
  set(ctx, arg0, 442, x394);
  set(ctx, arg0, 443, x395);
  set(ctx, arg0, 444, x396);
  set(ctx, arg0, 445, x397);
  set(ctx, arg0, 446, x398);
  set(ctx, arg0, 447, x399);
  set(ctx, arg0, 448, x400);
  set(ctx, arg0, 449, x401);
  set(ctx, arg0, 450, x402);
  set(ctx, arg0, 451, x403);
  set(ctx, arg0, 452, x404);
  set(ctx, arg0, 453, x405);
  set(ctx, arg0, 454, x406);
  set(ctx, arg0, 455, x407);
  set(ctx, arg0, 456, x408);
  set(ctx, arg0, 457, x409);
  set(ctx, arg0, 458, x410);
  set(ctx, arg0, 459, x411);
  set(ctx, arg0, 460, x412);
  set(ctx, arg0, 461, x413);
  set(ctx, arg0, 462, x414);
  set(ctx, arg0, 463, x415);
  set(ctx, arg0, 464, x343);
  set(ctx, arg0, 465, x344);
  set(ctx, arg0, 466, x345);
  set(ctx, arg0, 467, x346);
  set(ctx, arg0, 468, x347);
  set(ctx, arg0, 469, x348);
  set(ctx, arg0, 470, x349);
  set(ctx, arg0, 471, x350);
  set(ctx, arg0, 472, x351);
  set(ctx, arg0, 473, x352);
  set(ctx, arg0, 474, x353);
  set(ctx, arg0, 475, x354);
  set(ctx, arg0, 476, x355);
  set(ctx, arg0, 477, x356);
  set(ctx, arg0, 478, x357);
  set(ctx, arg0, 479, x358);
  set(ctx, arg0, 480, x359);
  set(ctx, arg0, 481, x360);
  set(ctx, arg0, 482, x361);
  set(ctx, arg0, 483, x362);
  set(ctx, arg0, 484, x363);
  set(ctx, arg0, 485, x364);
  set(ctx, arg0, 486, x365);
  set(ctx, arg0, 487, x366);
  set(ctx, arg0, 488, x367);
  set(ctx, arg0, 489, x368);
  set(ctx, arg0, 490, x369);
  set(ctx, arg0, 491, x370);
  set(ctx, arg0, 492, x371);
  set(ctx, arg0, 493, x372);
  set(ctx, arg0, 494, x373);
  set(ctx, arg0, 495, x374);
  set(ctx, arg0, 496, x302);
  set(ctx, arg0, 497, x303);
  set(ctx, arg0, 498, x304);
  set(ctx, arg0, 499, x305);
  set(ctx, arg0, 500, x306);
  set(ctx, arg0, 501, x307);
  set(ctx, arg0, 502, x308);
  set(ctx, arg0, 503, x309);
  set(ctx, arg0, 504, x310);
  set(ctx, arg0, 505, x311);
  set(ctx, arg0, 506, x312);
  set(ctx, arg0, 507, x313);
  set(ctx, arg0, 508, x314);
  set(ctx, arg0, 509, x315);
  set(ctx, arg0, 510, x316);
  set(ctx, arg0, 511, x317);
  set(ctx, arg0, 512, x318);
  set(ctx, arg0, 513, x319);
  set(ctx, arg0, 514, x320);
  set(ctx, arg0, 515, x321);
  set(ctx, arg0, 516, x322);
  set(ctx, arg0, 517, x323);
  set(ctx, arg0, 518, x324);
  set(ctx, arg0, 519, x325);
  set(ctx, arg0, 520, x326);
  set(ctx, arg0, 521, x327);
  set(ctx, arg0, 522, x328);
  set(ctx, arg0, 523, x329);
  set(ctx, arg0, 524, x330);
  set(ctx, arg0, 525, x331);
  set(ctx, arg0, 526, x332);
  set(ctx, arg0, 527, x333);
  set(ctx, arg0, 528, Val(0));
  set(ctx, arg0, 529, Val(0));
  set(ctx, arg0, 530, Val(0));
  set(ctx, arg0, 531, Val(0));
  set(ctx, arg0, 532, Val(0));
  set(ctx, arg0, 533, Val(0));
  set(ctx, arg0, 534, Val(0));
  set(ctx, arg0, 535, Val(0));
  set(ctx, arg0, 536, Val(0));
  set(ctx, arg0, 537, Val(0));
  set(ctx, arg0, 538, Val(0));
  set(ctx, arg0, 539, Val(0));
  set(ctx, arg0, 540, Val(0));
  set(ctx, arg0, 541, Val(0));
  set(ctx, arg0, 542, Val(0));
  set(ctx, arg0, 543, Val(0));
  set(ctx, arg0, 544, Val(0));
  set(ctx, arg0, 545, Val(0));
  set(ctx, arg0, 546, Val(0));
  set(ctx, arg0, 547, Val(0));
  set(ctx, arg0, 548, Val(0));
  set(ctx, arg0, 549, Val(0));
  set(ctx, arg0, 550, Val(0));
  set(ctx, arg0, 551, Val(0));
  set(ctx, arg0, 552, Val(0));
  set(ctx, arg0, 553, Val(0));
  set(ctx, arg0, 554, Val(0));
  set(ctx, arg0, 555, Val(0));
  set(ctx, arg0, 556, Val(0));
  set(ctx, arg0, 557, Val(0));
  set(ctx, arg0, 558, Val(0));
  set(ctx, arg0, 559, Val(0));
  set(ctx, arg0, 560, Val(0));
  set(ctx, arg0, 561, Val(0));
  set(ctx, arg0, 562, Val(0));
  set(ctx, arg0, 563, Val(0));
  set(ctx, arg0, 564, Val(0));
  set(ctx, arg0, 565, Val(0));
  set(ctx, arg0, 566, Val(0));
  set(ctx, arg0, 567, Val(0));
  set(ctx, arg0, 568, Val(0));
  set(ctx, arg0, 569, Val(0));
  set(ctx, arg0, 570, Val(0));
  set(ctx, arg0, 571, Val(0));
  set(ctx, arg0, 572, Val(0));
  set(ctx, arg0, 573, Val(0));
  set(ctx, arg0, 574, Val(0));
  set(ctx, arg0, 575, Val(0));
  set(ctx, arg0, 576, Val(0));
  set(ctx, arg0, 577, Val(0));
  set(ctx, arg0, 578, Val(0));
  set(ctx, arg0, 579, Val(0));
  set(ctx, arg0, 580, Val(0));
  set(ctx, arg0, 581, Val(0));
  set(ctx, arg0, 582, Val(0));
  set(ctx, arg0, 583, Val(0));
  set(ctx, arg0, 584, Val(0));
  set(ctx, arg0, 585, Val(0));
  set(ctx, arg0, 586, Val(0));
  set(ctx, arg0, 587, Val(0));
  set(ctx, arg0, 588, Val(0));
  set(ctx, arg0, 589, Val(0));
  set(ctx, arg0, 590, Val(0));
  set(ctx, arg0, 591, Val(0));
  set(ctx, arg0, 592, Val(0));
  set(ctx, arg0, 593, Val(0));
  set(ctx, arg0, 594, Val(0));
  set(ctx, arg0, 595, Val(0));
  set(ctx, arg0, 596, Val(0));
  set(ctx, arg0, 597, Val(0));
  set(ctx, arg0, 598, Val(0));
  set(ctx, arg0, 599, Val(0));
  set(ctx, arg0, 600, Val(0));
  set(ctx, arg0, 601, Val(0));
  set(ctx, arg0, 602, Val(0));
  set(ctx, arg0, 603, Val(0));
  set(ctx, arg0, 604, Val(0));
  set(ctx, arg0, 605, Val(0));
  set(ctx, arg0, 606, Val(0));
  set(ctx, arg0, 607, Val(0));
  set(ctx, arg0, 608, Val(0));
  set(ctx, arg0, 609, Val(0));
  set(ctx, arg0, 610, Val(0));
  set(ctx, arg0, 611, Val(0));
  set(ctx, arg0, 612, Val(0));
  set(ctx, arg0, 613, Val(0));
  set(ctx, arg0, 614, Val(0));
  set(ctx, arg0, 615, Val(0));
  set(ctx, arg0, 616, Val(0));
  set(ctx, arg0, 617, Val(0));
  set(ctx, arg0, 618, Val(0));
  set(ctx, arg0, 619, Val(0));
  set(ctx, arg0, 620, Val(0));
  set(ctx, arg0, 621, Val(0));
  set(ctx, arg0, 622, Val(0));
  set(ctx, arg0, 623, Val(0));
  set(ctx, arg0, 624, Val(0));
  set(ctx, arg0, 625, Val(0));
  set(ctx, arg0, 626, Val(0));
  set(ctx, arg0, 627, Val(0));
  set(ctx, arg0, 628, Val(0));
  set(ctx, arg0, 629, Val(0));
  set(ctx, arg0, 630, Val(0));
  set(ctx, arg0, 631, Val(0));
  set(ctx, arg0, 632, Val(0));
  set(ctx, arg0, 633, Val(0));
  set(ctx, arg0, 634, Val(0));
  set(ctx, arg0, 635, Val(0));
  set(ctx, arg0, 636, Val(0));
  set(ctx, arg0, 637, Val(0));
  set(ctx, arg0, 638, Val(0));
  set(ctx, arg0, 639, Val(0));
  set(ctx, arg0, 640, Val(0));
  set(ctx, arg0, 641, Val(0));
  set(ctx, arg0, 642, Val(0));
  set(ctx, arg0, 643, Val(0));
  set(ctx, arg0, 644, Val(0));
  set(ctx, arg0, 645, Val(0));
  set(ctx, arg0, 646, Val(0));
  set(ctx, arg0, 647, Val(0));
  set(ctx, arg0, 648, Val(0));
  set(ctx, arg0, 649, Val(0));
  set(ctx, arg0, 650, Val(0));
  set(ctx, arg0, 651, Val(0));
  set(ctx, arg0, 652, Val(0));
  set(ctx, arg0, 653, Val(0));
  set(ctx, arg0, 654, Val(0));
  set(ctx, arg0, 655, Val(0));
  set(ctx, arg0, 656, Val(0));
  set(ctx, arg0, 657, Val(0));
  set(ctx, arg0, 658, Val(0));
  set(ctx, arg0, 659, Val(0));
  set(ctx, arg0, 660, Val(0));
  set(ctx, arg0, 661, Val(0));
  set(ctx, arg0, 662, Val(0));
  set(ctx, arg0, 663, Val(0));
  set(ctx, arg0, 664, Val(0));
  set(ctx, arg0, 665, Val(0));
  set(ctx, arg0, 666, Val(0));
  set(ctx, arg0, 667, Val(0));
  set(ctx, arg0, 668, Val(0));
  set(ctx, arg0, 669, Val(0));
  set(ctx, arg0, 670, Val(0));
  set(ctx, arg0, 671, Val(0));
  set(ctx, arg0, 672, Val(0));
  set(ctx, arg0, 673, Val(0));
  set(ctx, arg0, 674, Val(0));
  set(ctx, arg0, 675, Val(0));
  set(ctx, arg0, 676, Val(0));
  set(ctx, arg0, 677, Val(0));
  set(ctx, arg0, 678, Val(0));
  set(ctx, arg0, 679, Val(0));
  set(ctx, arg0, 680, Val(0));
  set(ctx, arg0, 681, Val(0));
  set(ctx, arg0, 682, Val(0));
  set(ctx, arg0, 683, Val(0));
  set(ctx, arg0, 684, Val(0));
  set(ctx, arg0, 685, Val(0));
  set(ctx, arg0, 686, Val(0));
  set(ctx, arg0, 687, Val(0));
  set(ctx, arg0, 688, Val(0));
  set(ctx, arg0, 689, Val(0));
  set(ctx, arg0, 690, Val(0));
  set(ctx, arg0, 691, Val(0));
  set(ctx, arg0, 692, Val(0));
  set(ctx, arg0, 693, Val(0));
  set(ctx, arg0, 694, Val(0));
  set(ctx, arg0, 695, Val(0));
  set(ctx, arg0, 696, Val(0));
  set(ctx, arg0, 697, Val(0));
  set(ctx, arg0, 698, Val(0));
  set(ctx, arg0, 699, Val(0));
  set(ctx, arg0, 700, Val(0));
  set(ctx, arg0, 701, Val(0));
  set(ctx, arg0, 702, Val(0));
  set(ctx, arg0, 703, Val(0));
  set(ctx, arg0, 704, Val(0));
  set(ctx, arg0, 705, Val(0));
  set(ctx, arg0, 706, Val(0));
  set(ctx, arg0, 707, Val(0));
  set(ctx, arg0, 708, Val(0));
  set(ctx, arg0, 709, Val(0));
  set(ctx, arg0, 710, Val(0));
  set(ctx, arg0, 711, Val(0));
  set(ctx, arg0, 712, Val(0));
  set(ctx, arg0, 713, Val(0));
  set(ctx, arg0, 714, Val(0));
  set(ctx, arg0, 715, Val(0));
  set(ctx, arg0, 716, Val(0));
  set(ctx, arg0, 717, Val(0));
  set(ctx, arg0, 718, Val(0));
  set(ctx, arg0, 719, Val(0));
  set(ctx, arg0, 720, Val(0));
  set(ctx, arg0, 721, Val(0));
  set(ctx, arg0, 722, Val(0));
  set(ctx, arg0, 723, Val(0));
  set(ctx, arg0, 724, Val(0));
  set(ctx, arg0, 725, Val(0));
  set(ctx, arg0, 726, Val(0));
  set(ctx, arg0, 727, Val(0));
  set(ctx, arg0, 728, Val(0));
  set(ctx, arg0, 729, Val(0));
  set(ctx, arg0, 730, Val(0));
  set(ctx, arg0, 731, Val(0));
  set(ctx, arg0, 732, Val(0));
  set(ctx, arg0, 733, Val(0));
  set(ctx, arg0, 734, Val(0));
  set(ctx, arg0, 735, Val(0));
  set(ctx, arg0, 736, Val(0));
  set(ctx, arg0, 737, Val(0));
  set(ctx, arg0, 738, Val(0));
  set(ctx, arg0, 739, Val(0));
  set(ctx, arg0, 740, Val(0));
  set(ctx, arg0, 741, Val(0));
  set(ctx, arg0, 742, Val(0));
  set(ctx, arg0, 743, Val(0));
  set(ctx, arg0, 744, Val(0));
  set(ctx, arg0, 745, Val(0));
  set(ctx, arg0, 746, Val(0));
  set(ctx, arg0, 747, Val(0));
  set(ctx, arg0, 748, Val(0));
  set(ctx, arg0, 749, Val(0));
  set(ctx, arg0, 750, Val(0));
  set(ctx, arg0, 751, Val(0));
  set(ctx, arg0, 752, Val(0));
  set(ctx, arg0, 753, Val(0));
  set(ctx, arg0, 754, Val(0));
  set(ctx, arg0, 755, Val(0));
  set(ctx, arg0, 756, Val(0));
  set(ctx, arg0, 757, Val(0));
  set(ctx, arg0, 758, Val(0));
  set(ctx, arg0, 759, Val(0));
  set(ctx, arg0, 760, Val(0));
  set(ctx, arg0, 761, Val(0));
  set(ctx, arg0, 762, Val(0));
  set(ctx, arg0, 763, Val(0));
  set(ctx, arg0, 764, Val(0));
  set(ctx, arg0, 765, Val(0));
  set(ctx, arg0, 766, Val(0));
  set(ctx, arg0, 767, Val(0));
  set(ctx, arg0, 768, Val(0));
  set(ctx, arg0, 769, Val(0));
  set(ctx, arg0, 770, Val(0));
  set(ctx, arg0, 771, Val(0));
  set(ctx, arg0, 772, Val(0));
  set(ctx, arg0, 773, Val(0));
  set(ctx, arg0, 774, Val(0));
  set(ctx, arg0, 775, Val(0));
  set(ctx, arg0, 776, Val(0));
  set(ctx, arg0, 777, Val(0));
  set(ctx, arg0, 778, Val(0));
  set(ctx, arg0, 779, Val(0));
  set(ctx, arg0, 780, Val(0));
  set(ctx, arg0, 781, Val(0));
  set(ctx, arg0, 782, Val(0));
  set(ctx, arg0, 783, Val(0));
  set(ctx, arg0, 784, Val(0));
  set(ctx, arg0, 785, Val(0));
  set(ctx, arg0, 786, Val(0));
  set(ctx, arg0, 787, Val(0));
  set(ctx, arg0, 788, Val(0));
  set(ctx, arg0, 789, Val(0));
  set(ctx, arg0, 790, Val(0));
  set(ctx, arg0, 791, Val(0));
  set(ctx, arg0, 792, Val(0));
  set(ctx, arg0, 793, Val(0));
  set(ctx, arg0, 794, Val(0));
  set(ctx, arg0, 795, Val(0));
  set(ctx, arg0, 796, Val(0));
  set(ctx, arg0, 797, Val(0));
  set(ctx, arg0, 798, Val(0));
  set(ctx, arg0, 799, Val(0));
  set(ctx, arg0, 800, Val(0));
  set(ctx, arg0, 801, Val(0));
  set(ctx, arg0, 802, Val(0));
  set(ctx, arg0, 803, Val(0));
  set(ctx, arg0, 804, Val(0));
  set(ctx, arg0, 805, Val(0));
  set(ctx, arg0, 806, Val(0));
  set(ctx, arg0, 807, Val(0));
  set(ctx, arg0, 808, Val(0));
  set(ctx, arg0, 809, Val(0));
  set(ctx, arg0, 810, Val(0));
  set(ctx, arg0, 811, Val(0));
  set(ctx, arg0, 812, Val(0));
  set(ctx, arg0, 813, Val(0));
  set(ctx, arg0, 814, Val(0));
  set(ctx, arg0, 815, Val(0));
  // TopState(zirgen/circuit/keccak2/top.zir:40)
  set(ctx, arg0, 816, get(ctx, arg0, 816, 1));
  set(ctx, arg0, 817, get(ctx, arg0, 817, 1));
  set(ctx, arg0, 818, get(ctx, arg0, 818, 1));
  set(ctx, arg0, 819, get(ctx, arg0, 819, 1));
  set(ctx, arg0, 820, get(ctx, arg0, 820, 1));
  set(ctx, arg0, 821, get(ctx, arg0, 821, 1));
  set(ctx, arg0, 822, get(ctx, arg0, 822, 1));
  set(ctx, arg0, 823, get(ctx, arg0, 823, 1));
  set(ctx, arg0, 824, get(ctx, arg0, 824, 1));
  set(ctx, arg0, 825, get(ctx, arg0, 825, 1));
  set(ctx, arg0, 826, get(ctx, arg0, 826, 1));
  set(ctx, arg0, 827, get(ctx, arg0, 827, 1));
  set(ctx, arg0, 828, get(ctx, arg0, 828, 1));
  set(ctx, arg0, 829, get(ctx, arg0, 829, 1));
  set(ctx, arg0, 830, get(ctx, arg0, 830, 1));
  set(ctx, arg0, 831, get(ctx, arg0, 831, 1));
  set(ctx, arg0, 832, get(ctx, arg0, 832, 1));
  set(ctx, arg0, 833, get(ctx, arg0, 833, 1));
  set(ctx, arg0, 834, get(ctx, arg0, 834, 1));
  set(ctx, arg0, 835, get(ctx, arg0, 835, 1));
  set(ctx, arg0, 836, get(ctx, arg0, 836, 1));
  set(ctx, arg0, 837, get(ctx, arg0, 837, 1));
  set(ctx, arg0, 838, get(ctx, arg0, 838, 1));
  set(ctx, arg0, 839, get(ctx, arg0, 839, 1));
  set(ctx, arg0, 840, get(ctx, arg0, 840, 1));
  set(ctx, arg0, 841, get(ctx, arg0, 841, 1));
  set(ctx, arg0, 842, get(ctx, arg0, 842, 1));
  set(ctx, arg0, 843, get(ctx, arg0, 843, 1));
  set(ctx, arg0, 844, get(ctx, arg0, 844, 1));
  set(ctx, arg0, 845, get(ctx, arg0, 845, 1));
  set(ctx, arg0, 846, get(ctx, arg0, 846, 1));
  set(ctx, arg0, 847, get(ctx, arg0, 847, 1));
  set(ctx, arg0, 848, get(ctx, arg0, 848, 1));
  set(ctx, arg0, 849, get(ctx, arg0, 849, 1));
  set(ctx, arg0, 850, get(ctx, arg0, 850, 1));
  set(ctx, arg0, 851, get(ctx, arg0, 851, 1));
  set(ctx, arg0, 852, get(ctx, arg0, 852, 1));
  set(ctx, arg0, 853, get(ctx, arg0, 853, 1));
  set(ctx, arg0, 854, get(ctx, arg0, 854, 1));
  set(ctx, arg0, 855, get(ctx, arg0, 855, 1));
  set(ctx, arg0, 856, get(ctx, arg0, 856, 1));
  set(ctx, arg0, 857, get(ctx, arg0, 857, 1));
  set(ctx, arg0, 858, get(ctx, arg0, 858, 1));
  set(ctx, arg0, 859, get(ctx, arg0, 859, 1));
  set(ctx, arg0, 860, get(ctx, arg0, 860, 1));
  set(ctx, arg0, 861, get(ctx, arg0, 861, 1));
  set(ctx, arg0, 862, get(ctx, arg0, 862, 1));
  set(ctx, arg0, 863, get(ctx, arg0, 863, 1));
  set(ctx, arg0, 864, get(ctx, arg0, 864, 1));
  set(ctx, arg0, 865, get(ctx, arg0, 865, 1));
  set(ctx, arg0, 866, get(ctx, arg0, 866, 1));
  set(ctx, arg0, 867, get(ctx, arg0, 867, 1));
  set(ctx, arg0, 868, get(ctx, arg0, 868, 1));
  set(ctx, arg0, 869, get(ctx, arg0, 869, 1));
  set(ctx, arg0, 870, get(ctx, arg0, 870, 1));
  set(ctx, arg0, 871, get(ctx, arg0, 871, 1));
  set(ctx, arg0, 872, get(ctx, arg0, 872, 1));
  set(ctx, arg0, 873, get(ctx, arg0, 873, 1));
  set(ctx, arg0, 874, get(ctx, arg0, 874, 1));
  set(ctx, arg0, 875, get(ctx, arg0, 875, 1));
  set(ctx, arg0, 876, get(ctx, arg0, 876, 1));
  set(ctx, arg0, 877, get(ctx, arg0, 877, 1));
  set(ctx, arg0, 878, get(ctx, arg0, 878, 1));
  set(ctx, arg0, 879, get(ctx, arg0, 879, 1));
  set(ctx, arg0, 880, get(ctx, arg0, 880, 1));
  set(ctx, arg0, 881, get(ctx, arg0, 881, 1));
  set(ctx, arg0, 882, get(ctx, arg0, 882, 1));
  set(ctx, arg0, 883, get(ctx, arg0, 883, 1));
  set(ctx, arg0, 884, get(ctx, arg0, 884, 1));
  set(ctx, arg0, 885, get(ctx, arg0, 885, 1));
  set(ctx, arg0, 886, get(ctx, arg0, 886, 1));
  set(ctx, arg0, 887, get(ctx, arg0, 887, 1));
  set(ctx, arg0, 888, get(ctx, arg0, 888, 1));
  set(ctx, arg0, 889, get(ctx, arg0, 889, 1));
  set(ctx, arg0, 890, get(ctx, arg0, 890, 1));
  set(ctx, arg0, 891, get(ctx, arg0, 891, 1));
  set(ctx, arg0, 892, get(ctx, arg0, 892, 1));
  set(ctx, arg0, 893, get(ctx, arg0, 893, 1));
  set(ctx, arg0, 894, get(ctx, arg0, 894, 1));
  set(ctx, arg0, 895, get(ctx, arg0, 895, 1));
  set(ctx, arg0, 896, get(ctx, arg0, 896, 1));
  set(ctx, arg0, 897, get(ctx, arg0, 897, 1));
  set(ctx, arg0, 898, get(ctx, arg0, 898, 1));
  set(ctx, arg0, 899, get(ctx, arg0, 899, 1));
  set(ctx, arg0, 900, get(ctx, arg0, 900, 1));
  set(ctx, arg0, 901, get(ctx, arg0, 901, 1));
  set(ctx, arg0, 902, get(ctx, arg0, 902, 1));
  set(ctx, arg0, 903, get(ctx, arg0, 903, 1));
  set(ctx, arg0, 904, get(ctx, arg0, 904, 1));
  set(ctx, arg0, 905, get(ctx, arg0, 905, 1));
  set(ctx, arg0, 906, get(ctx, arg0, 906, 1));
  set(ctx, arg0, 907, get(ctx, arg0, 907, 1));
  set(ctx, arg0, 908, get(ctx, arg0, 908, 1));
  set(ctx, arg0, 909, get(ctx, arg0, 909, 1));
  set(ctx, arg0, 910, get(ctx, arg0, 910, 1));
  set(ctx, arg0, 911, get(ctx, arg0, 911, 1));
  set(ctx, arg0, 912, get(ctx, arg0, 912, 1));
  set(ctx, arg0, 913, get(ctx, arg0, 913, 1));
  set(ctx, arg0, 914, get(ctx, arg0, 914, 1));
  set(ctx, arg0, 915, get(ctx, arg0, 915, 1));
  // TopState(zirgen/circuit/keccak2/top.zir:43)
  set(ctx, arg0, 916, x461);
  set(ctx, arg0, 917, x466);
  set(ctx, arg0, 918, x471);
  set(ctx, arg0, 919, x476);
  set(ctx, arg0, 920, x481);
  set(ctx, arg0, 921, x486);
  set(ctx, arg0, 922, x491);
  set(ctx, arg0, 923, x496);
  set(ctx, arg0, 924, x501);
  set(ctx, arg0, 925, x506);
  set(ctx, arg0, 926, x511);
  set(ctx, arg0, 927, x516);
  set(ctx, arg0, 928, x521);
  set(ctx, arg0, 929, x526);
  set(ctx, arg0, 930, x531);
  set(ctx, arg0, 931, x536);
  return;
=======
__device__ NondetBitRegStruct back_NondetBitReg(ExecContext& ctx,
                                                Index distance0,
                                                BoundLayout<NondetRegLayout> layout1) {
  // NondetBitReg(zirgen/circuit/keccak2/bits.zir:11)
  NondetRegStruct x2 = back_NondetReg(ctx, distance0, layout1);
  return NondetBitRegStruct{._super = x2};
}
__device__ DoShaStepStruct exec_DoShaStep(ExecContext& ctx,
                                          ShaStateStruct arg0,
                                          Val2Array arg1,
                                          Val arg2,
                                          Val2Array arg3,
                                          BoundLayout<DoShaStepLayout> layout4) {
  // BitXor(zirgen/circuit/keccak2/bits.zir:32)
  // XorU32(zirgen/circuit/keccak2/sha2.zir:10)
  // ComputeW(zirgen/circuit/keccak2/sha2.zir:76)
  // DoShaStep(zirgen/circuit/keccak2/top.zir:137)
  Val x5 = (arg0.w[14][18] + arg0.w[14][3]);
  Val x6 = (arg0.w[14][18] * Val(2));
  Val x7 = (x5 - (x6 * arg0.w[14][3]));
  Val x8 = (arg0.w[14][19] + arg0.w[14][4]);
  Val x9 = (arg0.w[14][19] * Val(2));
  Val x10 = (x8 - (x9 * arg0.w[14][4]));
  Val x11 = (arg0.w[14][20] + arg0.w[14][5]);
  Val x12 = (arg0.w[14][20] * Val(2));
  Val x13 = (x11 - (x12 * arg0.w[14][5]));
  Val x14 = (arg0.w[14][21] + arg0.w[14][6]);
  Val x15 = (arg0.w[14][21] * Val(2));
  Val x16 = (x14 - (x15 * arg0.w[14][6]));
  Val x17 = (arg0.w[14][22] + arg0.w[14][7]);
  Val x18 = (arg0.w[14][22] * Val(2));
  Val x19 = (x17 - (x18 * arg0.w[14][7]));
  Val x20 = (arg0.w[14][23] + arg0.w[14][8]);
  Val x21 = (arg0.w[14][23] * Val(2));
  Val x22 = (x20 - (x21 * arg0.w[14][8]));
  Val x23 = (arg0.w[14][24] + arg0.w[14][9]);
  Val x24 = (arg0.w[14][24] * Val(2));
  Val x25 = (x23 - (x24 * arg0.w[14][9]));
  Val x26 = (arg0.w[14][25] + arg0.w[14][10]);
  Val x27 = (arg0.w[14][25] * Val(2));
  Val x28 = (x26 - (x27 * arg0.w[14][10]));
  Val x29 = (arg0.w[14][26] + arg0.w[14][11]);
  Val x30 = (arg0.w[14][26] * Val(2));
  Val x31 = (x29 - (x30 * arg0.w[14][11]));
  Val x32 = (arg0.w[14][27] + arg0.w[14][12]);
  Val x33 = (arg0.w[14][27] * Val(2));
  Val x34 = (x32 - (x33 * arg0.w[14][12]));
  Val x35 = (arg0.w[14][28] + arg0.w[14][13]);
  Val x36 = (arg0.w[14][28] * Val(2));
  Val x37 = (x35 - (x36 * arg0.w[14][13]));
  Val x38 = (arg0.w[14][29] + arg0.w[14][14]);
  Val x39 = (arg0.w[14][29] * Val(2));
  Val x40 = (x38 - (x39 * arg0.w[14][14]));
  Val x41 = (arg0.w[14][30] + arg0.w[14][15]);
  Val x42 = (arg0.w[14][30] * Val(2));
  Val x43 = (x41 - (x42 * arg0.w[14][15]));
  Val x44 = (arg0.w[14][31] + arg0.w[14][16]);
  Val x45 = (arg0.w[14][31] * Val(2));
  Val x46 = (x44 - (x45 * arg0.w[14][16]));
  Val x47 = (arg0.w[14][0] + arg0.w[14][17]);
  Val x48 = (arg0.w[14][0] * Val(2));
  Val x49 = (x47 - (x48 * arg0.w[14][17]));
  Val x50 = (arg0.w[14][1] + arg0.w[14][18]);
  Val x51 = (arg0.w[14][1] * Val(2));
  Val x52 = (x50 - (x51 * arg0.w[14][18]));
  Val x53 = (arg0.w[14][2] + arg0.w[14][19]);
  Val x54 = (arg0.w[14][2] * Val(2));
  Val x55 = (x53 - (x54 * arg0.w[14][19]));
  Val x56 = (arg0.w[14][3] + arg0.w[14][20]);
  Val x57 = (arg0.w[14][3] * Val(2));
  Val x58 = (x56 - (x57 * arg0.w[14][20]));
  Val x59 = (arg0.w[14][4] + arg0.w[14][21]);
  Val x60 = (arg0.w[14][4] * Val(2));
  Val x61 = (x59 - (x60 * arg0.w[14][21]));
  Val x62 = (arg0.w[14][5] + arg0.w[14][22]);
  Val x63 = (arg0.w[14][5] * Val(2));
  Val x64 = (x62 - (x63 * arg0.w[14][22]));
  Val x65 = (arg0.w[14][6] + arg0.w[14][23]);
  Val x66 = (arg0.w[14][6] * Val(2));
  Val x67 = (x65 - (x66 * arg0.w[14][23]));
  Val x68 = (arg0.w[14][7] + arg0.w[14][24]);
  Val x69 = (arg0.w[14][7] * Val(2));
  Val x70 = (x68 - (x69 * arg0.w[14][24]));
  Val x71 = (arg0.w[14][8] + arg0.w[14][25]);
  Val x72 = (arg0.w[14][8] * Val(2));
  Val x73 = (x71 - (x72 * arg0.w[14][25]));
  Val x74 = (arg0.w[14][9] + arg0.w[14][26]);
  Val x75 = (arg0.w[14][9] * Val(2));
  Val x76 = (x74 - (x75 * arg0.w[14][26]));
  Val x77 = (arg0.w[14][10] + arg0.w[14][27]);
  Val x78 = (arg0.w[14][10] * Val(2));
  Val x79 = (x77 - (x78 * arg0.w[14][27]));
  Val x80 = (arg0.w[14][11] + arg0.w[14][28]);
  Val x81 = (arg0.w[14][11] * Val(2));
  Val x82 = (x80 - (x81 * arg0.w[14][28]));
  Val x83 = (arg0.w[14][12] + arg0.w[14][29]);
  Val x84 = (arg0.w[14][12] * Val(2));
  Val x85 = (x83 - (x84 * arg0.w[14][29]));
  Val x86 = (arg0.w[14][13] + arg0.w[14][30]);
  Val x87 = (arg0.w[14][13] * Val(2));
  Val x88 = (x86 - (x87 * arg0.w[14][30]));
  Val x89 = (arg0.w[14][14] + arg0.w[14][31]);
  Val x90 = (arg0.w[14][14] * Val(2));
  Val x91 = (x89 - (x90 * arg0.w[14][31]));
  Val x92 = (arg0.w[14][7] + x7);
  Val x93 = (arg0.w[14][8] + x10);
  Val x94 = (arg0.w[14][9] + x13);
  Val x95 = (arg0.w[14][10] + x16);
  Val x96 = (arg0.w[14][11] + x19);
  Val x97 = (arg0.w[14][12] + x22);
  Val x98 = (arg0.w[14][13] + x25);
  Val x99 = (arg0.w[14][14] + x28);
  Val x100 = (arg0.w[14][15] + x31);
  Val x101 = (arg0.w[14][15] * Val(2));
  Val x102 = (arg0.w[14][16] + x34);
  Val x103 = (arg0.w[14][16] * Val(2));
  Val x104 = (arg0.w[14][17] + x37);
  Val x105 = (arg0.w[14][17] * Val(2));
  Val x106 = (arg0.w[14][18] + x40);
  Val x107 = (arg0.w[14][19] + x43);
  Val x108 = (arg0.w[14][20] + x46);
  Val x109 = (arg0.w[14][21] + x49);
  Val x110 = (arg0.w[14][22] + x52);
  Val x111 = (arg0.w[14][23] + x55);
  Val x112 = (arg0.w[14][24] + x58);
  Val x113 = (arg0.w[14][25] + x61);
  Val x114 = (arg0.w[14][26] + x64);
  Val x115 = (arg0.w[14][27] + x67);
  Val x116 = (arg0.w[14][28] + x70);
  Val x117 = (arg0.w[14][29] + x73);
  Val x118 = (arg0.w[14][30] + x76);
  Val x119 = (arg0.w[14][31] + x79);
  Val x120 = (arg0.w[14][0] + x82);
  Val x121 = (arg0.w[14][1] + x85);
  Val x122 = (arg0.w[14][2] + x88);
  Val x123 = (arg0.w[14][3] + x91);
  Val x124 = (arg0.w[14][4] + arg0.w[14][15]);
  Val x125 = (arg0.w[14][5] + arg0.w[14][16]);
  Val x126 = (arg0.w[14][6] + arg0.w[14][17]);
  // ComputeW(zirgen/circuit/keccak2/sha2.zir:77)
  Val x127 = (arg0.w[1][19] + arg0.w[1][10]);
  Val x128 = (arg0.w[1][19] * Val(2));
  Val x129 = (x127 - (x128 * arg0.w[1][10]));
  Val x130 = (arg0.w[1][20] + arg0.w[1][11]);
  Val x131 = (arg0.w[1][20] * Val(2));
  Val x132 = (x130 - (x131 * arg0.w[1][11]));
  Val x133 = (arg0.w[1][21] + arg0.w[1][12]);
  Val x134 = (arg0.w[1][21] * Val(2));
  Val x135 = (x133 - (x134 * arg0.w[1][12]));
  Val x136 = (arg0.w[1][22] + arg0.w[1][13]);
  Val x137 = (arg0.w[1][22] * Val(2));
  Val x138 = (x136 - (x137 * arg0.w[1][13]));
  Val x139 = (arg0.w[1][23] + arg0.w[1][14]);
  Val x140 = (arg0.w[1][23] * Val(2));
  Val x141 = (x139 - (x140 * arg0.w[1][14]));
  Val x142 = (arg0.w[1][24] + arg0.w[1][15]);
  Val x143 = (arg0.w[1][24] * Val(2));
  Val x144 = (x142 - (x143 * arg0.w[1][15]));
  Val x145 = (arg0.w[1][25] + arg0.w[1][16]);
  Val x146 = (arg0.w[1][25] * Val(2));
  Val x147 = (x145 - (x146 * arg0.w[1][16]));
  Val x148 = (arg0.w[1][26] + arg0.w[1][17]);
  Val x149 = (arg0.w[1][26] * Val(2));
  Val x150 = (x148 - (x149 * arg0.w[1][17]));
  Val x151 = (arg0.w[1][27] + arg0.w[1][18]);
  Val x152 = (arg0.w[1][27] * Val(2));
  Val x153 = (x151 - (x152 * arg0.w[1][18]));
  Val x154 = (arg0.w[1][28] + arg0.w[1][19]);
  Val x155 = (arg0.w[1][28] * Val(2));
  Val x156 = (x154 - (x155 * arg0.w[1][19]));
  Val x157 = (arg0.w[1][29] + arg0.w[1][20]);
  Val x158 = (arg0.w[1][29] * Val(2));
  Val x159 = (x157 - (x158 * arg0.w[1][20]));
  Val x160 = (arg0.w[1][30] + arg0.w[1][21]);
  Val x161 = (arg0.w[1][30] * Val(2));
  Val x162 = (x160 - (x161 * arg0.w[1][21]));
  Val x163 = (arg0.w[1][31] + arg0.w[1][22]);
  Val x164 = (arg0.w[1][31] * Val(2));
  Val x165 = (x163 - (x164 * arg0.w[1][22]));
  Val x166 = (arg0.w[1][0] + arg0.w[1][23]);
  Val x167 = (arg0.w[1][0] * Val(2));
  Val x168 = (x166 - (x167 * arg0.w[1][23]));
  Val x169 = (arg0.w[1][1] + arg0.w[1][24]);
  Val x170 = (arg0.w[1][1] * Val(2));
  Val x171 = (x169 - (x170 * arg0.w[1][24]));
  Val x172 = (arg0.w[1][2] + arg0.w[1][25]);
  Val x173 = (arg0.w[1][2] * Val(2));
  Val x174 = (x172 - (x173 * arg0.w[1][25]));
  Val x175 = (arg0.w[1][3] + arg0.w[1][26]);
  Val x176 = (arg0.w[1][3] * Val(2));
  Val x177 = (x175 - (x176 * arg0.w[1][26]));
  Val x178 = (arg0.w[1][4] + arg0.w[1][27]);
  Val x179 = (arg0.w[1][4] * Val(2));
  Val x180 = (x178 - (x179 * arg0.w[1][27]));
  Val x181 = (arg0.w[1][5] + arg0.w[1][28]);
  Val x182 = (arg0.w[1][5] * Val(2));
  Val x183 = (x181 - (x182 * arg0.w[1][28]));
  Val x184 = (arg0.w[1][6] + arg0.w[1][29]);
  Val x185 = (arg0.w[1][6] * Val(2));
  Val x186 = (x184 - (x185 * arg0.w[1][29]));
  Val x187 = (arg0.w[1][7] + arg0.w[1][30]);
  Val x188 = (arg0.w[1][7] * Val(2));
  Val x189 = (x187 - (x188 * arg0.w[1][30]));
  Val x190 = (arg0.w[1][8] + arg0.w[1][31]);
  Val x191 = (arg0.w[1][8] * Val(2));
  Val x192 = (x190 - (x191 * arg0.w[1][31]));
  Val x193 = (arg0.w[1][17] + x129);
  Val x194 = (arg0.w[1][17] * Val(2));
  Val x195 = (arg0.w[1][18] + x132);
  Val x196 = (arg0.w[1][18] * Val(2));
  Val x197 = (arg0.w[1][19] + x135);
  Val x198 = (arg0.w[1][20] + x138);
  Val x199 = (arg0.w[1][21] + x141);
  Val x200 = (arg0.w[1][22] + x144);
  Val x201 = (arg0.w[1][23] + x147);
  Val x202 = (arg0.w[1][24] + x150);
  Val x203 = (arg0.w[1][25] + x153);
  Val x204 = (arg0.w[1][26] + x156);
  Val x205 = (arg0.w[1][27] + x159);
  Val x206 = (arg0.w[1][28] + x162);
  Val x207 = (arg0.w[1][29] + x165);
  Val x208 = (arg0.w[1][30] + x168);
  Val x209 = (arg0.w[1][31] + x171);
  Val x210 = (arg0.w[1][0] + x174);
  Val x211 = (arg0.w[1][1] + x177);
  Val x212 = (arg0.w[1][2] + x180);
  Val x213 = (arg0.w[1][3] + x183);
  Val x214 = (arg0.w[1][4] + x186);
  Val x215 = (arg0.w[1][5] + x189);
  Val x216 = (arg0.w[1][6] + x192);
  Val x217 = (arg0.w[1][7] + arg0.w[1][9]);
  Val x218 = (arg0.w[1][8] + arg0.w[1][10]);
  Val x219 = (arg0.w[1][9] + arg0.w[1][11]);
  Val x220 = (arg0.w[1][9] * Val(2));
  Val x221 = (arg0.w[1][10] + arg0.w[1][12]);
  Val x222 = (arg0.w[1][10] * Val(2));
  Val x223 = (arg0.w[1][11] + arg0.w[1][13]);
  Val x224 = (arg0.w[1][11] * Val(2));
  Val x225 = (arg0.w[1][12] + arg0.w[1][14]);
  Val x226 = (arg0.w[1][12] * Val(2));
  Val x227 = (arg0.w[1][13] + arg0.w[1][15]);
  Val x228 = (arg0.w[1][13] * Val(2));
  Val x229 = (arg0.w[1][14] + arg0.w[1][16]);
  Val x230 = (arg0.w[1][14] * Val(2));
  Val x231 = (arg0.w[1][15] + arg0.w[1][17]);
  Val x232 = (arg0.w[1][15] * Val(2));
  Val x233 = (arg0.w[1][16] + arg0.w[1][18]);
  Val x234 = (arg0.w[1][16] * Val(2));
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:72)
  // ComputeW(zirgen/circuit/keccak2/sha2.zir:78)
  Val x235 = ((x92 - (x69 * x7)) + ((x93 - (x72 * x10)) * Val(2)));
  Val x236 = ((x235 + ((x94 - (x75 * x13)) * Val(4))) + ((x95 - (x78 * x16)) * Val(8)));
  Val x237 = ((x236 + ((x96 - (x81 * x19)) * Val(16))) + ((x97 - (x84 * x22)) * Val(32)));
  Val x238 = ((x237 + ((x98 - (x87 * x25)) * Val(64))) + ((x99 - (x90 * x28)) * Val(128)));
  Val x239 = ((x238 + ((x100 - (x101 * x31)) * Val(256))) + ((x102 - (x103 * x34)) * Val(512)));
  Val x240 = ((x239 + ((x104 - (x105 * x37)) * Val(1024))) + ((x106 - (x6 * x40)) * Val(2048)));
  Val x241 = ((x240 + ((x107 - (x9 * x43)) * Val(4096))) + ((x108 - (x12 * x46)) * Val(8192)));
  Val x242 = ((x241 + ((x109 - (x15 * x49)) * Val(16384))) + ((x110 - (x18 * x52)) * Val(32768)));
  Val x243 = ((x111 - (x21 * x55)) + ((x112 - (x24 * x58)) * Val(2)));
  Val x244 = ((x243 + ((x113 - (x27 * x61)) * Val(4))) + ((x114 - (x30 * x64)) * Val(8)));
  Val x245 = ((x244 + ((x115 - (x33 * x67)) * Val(16))) + ((x116 - (x36 * x70)) * Val(32)));
  Val x246 = ((x245 + ((x117 - (x39 * x73)) * Val(64))) + ((x118 - (x42 * x76)) * Val(128)));
  Val x247 = ((x246 + ((x119 - (x45 * x79)) * Val(256))) + ((x120 - (x48 * x82)) * Val(512)));
  Val x248 = ((x247 + ((x121 - (x51 * x85)) * Val(1024))) + ((x122 - (x54 * x88)) * Val(2048)));
  Val x249 =
      ((x248 + ((x123 - (x57 * x91)) * Val(4096))) + ((x124 - (x60 * arg0.w[14][15])) * Val(8192)));
  Val x250 = ((x249 + ((x125 - (x63 * arg0.w[14][16])) * Val(16384))) +
              ((x126 - (x66 * arg0.w[14][17])) * Val(32768)));
  Val x251 = ((x193 - (x194 * x129)) + ((x195 - (x196 * x132)) * Val(2)));
  Val x252 = ((x251 + ((x197 - (x128 * x135)) * Val(4))) + ((x198 - (x131 * x138)) * Val(8)));
  Val x253 = ((x252 + ((x199 - (x134 * x141)) * Val(16))) + ((x200 - (x137 * x144)) * Val(32)));
  Val x254 = ((x253 + ((x201 - (x140 * x147)) * Val(64))) + ((x202 - (x143 * x150)) * Val(128)));
  Val x255 = ((x254 + ((x203 - (x146 * x153)) * Val(256))) + ((x204 - (x149 * x156)) * Val(512)));
  Val x256 = ((x255 + ((x205 - (x152 * x159)) * Val(1024))) + ((x206 - (x155 * x162)) * Val(2048)));
  Val x257 = ((x256 + ((x207 - (x158 * x165)) * Val(4096))) + ((x208 - (x161 * x168)) * Val(8192)));
  Val x258 =
      ((x257 + ((x209 - (x164 * x171)) * Val(16384))) + ((x210 - (x167 * x174)) * Val(32768)));
  Val x259 = ((x211 - (x170 * x177)) + ((x212 - (x173 * x180)) * Val(2)));
  Val x260 = ((x259 + ((x213 - (x176 * x183)) * Val(4))) + ((x214 - (x179 * x186)) * Val(8)));
  Val x261 = ((x260 + ((x215 - (x182 * x189)) * Val(16))) + ((x216 - (x185 * x192)) * Val(32)));
  Val x262 = ((x261 + ((x217 - (x188 * arg0.w[1][9])) * Val(64))) +
              ((x218 - (x191 * arg0.w[1][10])) * Val(128)));
  Val x263 = ((x262 + ((x219 - (x220 * arg0.w[1][11])) * Val(256))) +
              ((x221 - (x222 * arg0.w[1][12])) * Val(512)));
  Val x264 = ((x263 + ((x223 - (x224 * arg0.w[1][13])) * Val(1024))) +
              ((x225 - (x226 * arg0.w[1][14])) * Val(2048)));
  Val x265 = ((x264 + ((x227 - (x228 * arg0.w[1][15])) * Val(4096))) +
              ((x229 - (x230 * arg0.w[1][16])) * Val(8192)));
  Val x266 = ((x265 + ((x231 - (x232 * arg0.w[1][17])) * Val(16384))) +
              ((x233 - (x234 * arg0.w[1][18])) * Val(32768)));
  Val x267 = (arg0.w[15][1] * Val(2));
  Val x268 = (arg0.w[15][2] * Val(4));
  Val x269 = (arg0.w[15][3] * Val(8));
  Val x270 = (arg0.w[15][4] * Val(16));
  Val x271 = (arg0.w[15][5] * Val(32));
  Val x272 = (arg0.w[15][6] * Val(64));
  Val x273 = (arg0.w[15][7] * Val(128));
  Val x274 = (arg0.w[15][8] * Val(256));
  Val x275 = (arg0.w[15][9] * Val(512));
  Val x276 = (arg0.w[15][10] * Val(1024));
  Val x277 = (arg0.w[15][11] * Val(2048));
  Val x278 = (arg0.w[15][12] * Val(4096));
  Val x279 = (arg0.w[15][13] * Val(8192));
  Val x280 = (arg0.w[15][14] * Val(16384));
  Val x281 = (arg0.w[15][15] * Val(32768));
  Val x282 = (arg0.w[15][0] + x267);
  Val x283 = (((x282 + x268) + x269) + x270);
  Val x284 = (((x283 + x271) + x272) + x273);
  Val x285 = (((x284 + x274) + x275) + x276);
  Val x286 = (((x285 + x277) + x278) + x279);
  Val x287 = (arg0.w[15][17] * Val(2));
  Val x288 = (arg0.w[15][18] * Val(4));
  Val x289 = (arg0.w[15][19] * Val(8));
  Val x290 = (arg0.w[15][20] * Val(16));
  Val x291 = (arg0.w[15][21] * Val(32));
  Val x292 = (arg0.w[15][22] * Val(64));
  Val x293 = (arg0.w[15][23] * Val(128));
  Val x294 = (arg0.w[15][24] * Val(256));
  Val x295 = (arg0.w[15][25] * Val(512));
  Val x296 = (arg0.w[15][26] * Val(1024));
  Val x297 = (arg0.w[15][27] * Val(2048));
  Val x298 = (arg0.w[15][28] * Val(4096));
  Val x299 = (arg0.w[15][29] * Val(8192));
  Val x300 = (arg0.w[15][30] * Val(16384));
  Val x301 = (arg0.w[15][31] * Val(32768));
  Val x302 = (arg0.w[15][16] + x287);
  Val x303 = (((x302 + x288) + x289) + x290);
  Val x304 = (((x303 + x291) + x292) + x293);
  Val x305 = (((x304 + x294) + x295) + x296);
  Val x306 = (((x305 + x297) + x298) + x299);
  Val x307 = (arg0.w[6][1] * Val(2));
  Val x308 = (arg0.w[6][2] * Val(4));
  Val x309 = (arg0.w[6][3] * Val(8));
  Val x310 = (arg0.w[6][4] * Val(16));
  Val x311 = (arg0.w[6][5] * Val(32));
  Val x312 = (arg0.w[6][6] * Val(64));
  Val x313 = (arg0.w[6][7] * Val(128));
  Val x314 = (arg0.w[6][8] * Val(256));
  Val x315 = (arg0.w[6][9] * Val(512));
  Val x316 = (arg0.w[6][10] * Val(1024));
  Val x317 = (arg0.w[6][11] * Val(2048));
  Val x318 = (arg0.w[6][12] * Val(4096));
  Val x319 = (arg0.w[6][13] * Val(8192));
  Val x320 = (arg0.w[6][14] * Val(16384));
  Val x321 = (arg0.w[6][15] * Val(32768));
  Val x322 = (arg0.w[6][0] + x307);
  Val x323 = (((x322 + x308) + x309) + x310);
  Val x324 = (((x323 + x311) + x312) + x313);
  Val x325 = (((x324 + x314) + x315) + x316);
  Val x326 = (((x325 + x317) + x318) + x319);
  Val x327 = (arg0.w[6][17] * Val(2));
  Val x328 = (arg0.w[6][18] * Val(4));
  Val x329 = (arg0.w[6][19] * Val(8));
  Val x330 = (arg0.w[6][20] * Val(16));
  Val x331 = (arg0.w[6][21] * Val(32));
  Val x332 = (arg0.w[6][22] * Val(64));
  Val x333 = (arg0.w[6][23] * Val(128));
  Val x334 = (arg0.w[6][24] * Val(256));
  Val x335 = (arg0.w[6][25] * Val(512));
  Val x336 = (arg0.w[6][26] * Val(1024));
  Val x337 = (arg0.w[6][27] * Val(2048));
  Val x338 = (arg0.w[6][28] * Val(4096));
  Val x339 = (arg0.w[6][29] * Val(8192));
  Val x340 = (arg0.w[6][30] * Val(16384));
  Val x341 = (arg0.w[6][31] * Val(32768));
  Val x342 = (arg0.w[6][16] + x327);
  Val x343 = (((x342 + x328) + x329) + x330);
  Val x344 = (((x343 + x331) + x332) + x333);
  Val x345 = (((x344 + x334) + x335) + x336);
  Val x346 = (((x345 + x337) + x338) + x339);
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x347 = (((x286 + x280) + x281) + ((x326 + x320) + x321));
  Val x348 = (((x306 + x300) + x301) + ((x346 + x340) + x341));
  NondetBitRegStruct32Array x349;
  if (to_size_t(arg2)) {
    // DoShaStep(zirgen/circuit/keccak2/top.zir:138)
    ExpandBEStruct x350 = exec_ExpandBE(ctx, arg3, LAYOUT_LOOKUP(layout4, w.arm0));
    NondetBitRegStruct32Array x351 = NondetBitRegStruct32Array{
        x350._super[0],  x350._super[1],  x350._super[2],  x350._super[3],  x350._super[4],
        x350._super[5],  x350._super[6],  x350._super[7],  x350._super[8],  x350._super[9],
        x350._super[10], x350._super[11], x350._super[12], x350._super[13], x350._super[14],
        x350._super[15], x350._super[16], x350._super[17], x350._super[18], x350._super[19],
        x350._super[20], x350._super[21], x350._super[22], x350._super[23], x350._super[24],
        x350._super[25], x350._super[26], x350._super[27], x350._super[28], x350._super[29],
        x350._super[30], x350._super[31]};
    x349 = x351;
  } else if (to_size_t((Val(1) - arg2))) {
    UnpackReg_32__16_Struct x352 =
        exec_CarryAndExpand(ctx,
                            Val2Array{(x242 + (x258 + x347)), (x250 + (x266 + x348))},
                            LAYOUT_LOOKUP(layout4, w.arm1));
    NondetBitRegStruct32Array x353 = NondetBitRegStruct32Array{
        x352._super[0]._super,  x352._super[1]._super,  x352._super[2]._super,
        x352._super[3]._super,  x352._super[4]._super,  x352._super[5]._super,
        x352._super[6]._super,  x352._super[7]._super,  x352._super[8]._super,
        x352._super[9]._super,  x352._super[10]._super, x352._super[11]._super,
        x352._super[12]._super, x352._super[13]._super, x352._super[14]._super,
        x352._super[15]._super, x352._super[16]._super, x352._super[17]._super,
        x352._super[18]._super, x352._super[19]._super, x352._super[20]._super,
        x352._super[21]._super, x352._super[22]._super, x352._super[23]._super,
        x352._super[24]._super, x352._super[25]._super, x352._super[26]._super,
        x352._super[27]._super, x352._super[28]._super, x352._super[29]._super,
        x352._super[30]._super, x352._super[31]._super};
    x349 = x353;
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  ComponentStruct x354 = ComponentStruct{};
  ComponentStruct32Array x355 =
      ComponentStruct32Array{x354, x354, x354, x354, x354, x354, x354, x354, x354, x354, x354,
                             x354, x354, x354, x354, x354, x354, x354, x354, x354, x354, x354,
                             x354, x354, x354, x354, x354, x354, x354, x354, x354, x354};
  NondetBitRegStruct32Array x356 =
      map(x355,
          LAYOUT_LOOKUP(layout4, w._super),
          ([&](ComponentStruct32Array::value_type x357,
               BoundLayout<NondetRegLayout32LayoutArray::value_type> x358) {
            NondetBitRegStruct x359 = back_NondetBitReg(ctx, 0, x358);
            return x359;
          }));
  NondetBitRegStruct32Array x360 =
      map(x355,
          LAYOUT_LOOKUP(layout4, w._super),
          ([&](ComponentStruct32Array::value_type x361,
               BoundLayout<NondetRegLayout32LayoutArray::value_type> x362) {
            NondetBitRegStruct x363 = back_NondetBitReg(ctx, 0, x362);
            return x363;
          }));
  // BitXor(zirgen/circuit/keccak2/bits.zir:32)
  // XorU32(zirgen/circuit/keccak2/sha2.zir:10)
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:89)
  // DoShaStep(zirgen/circuit/keccak2/top.zir:139)
  Val x364 = (arg0.a[0][13] + arg0.a[0][22]);
  Val x365 = (arg0.a[0][13] * Val(2));
  Val x366 = (x364 - (x365 * arg0.a[0][22]));
  Val x367 = (arg0.a[0][14] + arg0.a[0][23]);
  Val x368 = (arg0.a[0][14] * Val(2));
  Val x369 = (x367 - (x368 * arg0.a[0][23]));
  Val x370 = (arg0.a[0][15] + arg0.a[0][24]);
  Val x371 = (arg0.a[0][15] * Val(2));
  Val x372 = (x370 - (x371 * arg0.a[0][24]));
  Val x373 = (arg0.a[0][16] + arg0.a[0][25]);
  Val x374 = (arg0.a[0][16] * Val(2));
  Val x375 = (x373 - (x374 * arg0.a[0][25]));
  Val x376 = (arg0.a[0][17] + arg0.a[0][26]);
  Val x377 = (arg0.a[0][17] * Val(2));
  Val x378 = (x376 - (x377 * arg0.a[0][26]));
  Val x379 = (arg0.a[0][18] + arg0.a[0][27]);
  Val x380 = (arg0.a[0][18] * Val(2));
  Val x381 = (x379 - (x380 * arg0.a[0][27]));
  Val x382 = (arg0.a[0][19] + arg0.a[0][28]);
  Val x383 = (arg0.a[0][19] * Val(2));
  Val x384 = (x382 - (x383 * arg0.a[0][28]));
  Val x385 = (arg0.a[0][20] + arg0.a[0][29]);
  Val x386 = (arg0.a[0][20] * Val(2));
  Val x387 = (x385 - (x386 * arg0.a[0][29]));
  Val x388 = (arg0.a[0][21] + arg0.a[0][30]);
  Val x389 = (arg0.a[0][21] * Val(2));
  Val x390 = (x388 - (x389 * arg0.a[0][30]));
  Val x391 = (arg0.a[0][22] + arg0.a[0][31]);
  Val x392 = (arg0.a[0][22] * Val(2));
  Val x393 = (x391 - (x392 * arg0.a[0][31]));
  Val x394 = (arg0.a[0][23] + arg0.a[0][0]);
  Val x395 = (arg0.a[0][23] * Val(2));
  Val x396 = (x394 - (x395 * arg0.a[0][0]));
  Val x397 = (arg0.a[0][24] + arg0.a[0][1]);
  Val x398 = (arg0.a[0][24] * Val(2));
  Val x399 = (x397 - (x398 * arg0.a[0][1]));
  Val x400 = (arg0.a[0][25] + arg0.a[0][2]);
  Val x401 = (arg0.a[0][25] * Val(2));
  Val x402 = (x400 - (x401 * arg0.a[0][2]));
  Val x403 = (arg0.a[0][26] + arg0.a[0][3]);
  Val x404 = (arg0.a[0][26] * Val(2));
  Val x405 = (x403 - (x404 * arg0.a[0][3]));
  Val x406 = (arg0.a[0][27] + arg0.a[0][4]);
  Val x407 = (arg0.a[0][27] * Val(2));
  Val x408 = (x406 - (x407 * arg0.a[0][4]));
  Val x409 = (arg0.a[0][28] + arg0.a[0][5]);
  Val x410 = (arg0.a[0][28] * Val(2));
  Val x411 = (x409 - (x410 * arg0.a[0][5]));
  Val x412 = (arg0.a[0][29] + arg0.a[0][6]);
  Val x413 = (arg0.a[0][29] * Val(2));
  Val x414 = (x412 - (x413 * arg0.a[0][6]));
  Val x415 = (arg0.a[0][30] + arg0.a[0][7]);
  Val x416 = (arg0.a[0][30] * Val(2));
  Val x417 = (x415 - (x416 * arg0.a[0][7]));
  Val x418 = (arg0.a[0][31] + arg0.a[0][8]);
  Val x419 = (arg0.a[0][31] * Val(2));
  Val x420 = (x418 - (x419 * arg0.a[0][8]));
  Val x421 = (arg0.a[0][0] + arg0.a[0][9]);
  Val x422 = (arg0.a[0][0] * Val(2));
  Val x423 = (x421 - (x422 * arg0.a[0][9]));
  Val x424 = (arg0.a[0][1] + arg0.a[0][10]);
  Val x425 = (arg0.a[0][1] * Val(2));
  Val x426 = (x424 - (x425 * arg0.a[0][10]));
  Val x427 = (arg0.a[0][2] + arg0.a[0][11]);
  Val x428 = (arg0.a[0][2] * Val(2));
  Val x429 = (x427 - (x428 * arg0.a[0][11]));
  Val x430 = (arg0.a[0][3] + arg0.a[0][12]);
  Val x431 = (arg0.a[0][3] * Val(2));
  Val x432 = (x430 - (x431 * arg0.a[0][12]));
  Val x433 = (arg0.a[0][4] + arg0.a[0][13]);
  Val x434 = (arg0.a[0][4] * Val(2));
  Val x435 = (x433 - (x434 * arg0.a[0][13]));
  Val x436 = (arg0.a[0][5] + arg0.a[0][14]);
  Val x437 = (arg0.a[0][5] * Val(2));
  Val x438 = (x436 - (x437 * arg0.a[0][14]));
  Val x439 = (arg0.a[0][6] + arg0.a[0][15]);
  Val x440 = (arg0.a[0][6] * Val(2));
  Val x441 = (x439 - (x440 * arg0.a[0][15]));
  Val x442 = (arg0.a[0][7] + arg0.a[0][16]);
  Val x443 = (arg0.a[0][7] * Val(2));
  Val x444 = (x442 - (x443 * arg0.a[0][16]));
  Val x445 = (arg0.a[0][8] + arg0.a[0][17]);
  Val x446 = (arg0.a[0][8] * Val(2));
  Val x447 = (x445 - (x446 * arg0.a[0][17]));
  Val x448 = (arg0.a[0][9] + arg0.a[0][18]);
  Val x449 = (arg0.a[0][9] * Val(2));
  Val x450 = (x448 - (x449 * arg0.a[0][18]));
  Val x451 = (arg0.a[0][10] + arg0.a[0][19]);
  Val x452 = (arg0.a[0][10] * Val(2));
  Val x453 = (x451 - (x452 * arg0.a[0][19]));
  Val x454 = (arg0.a[0][11] + arg0.a[0][20]);
  Val x455 = (arg0.a[0][11] * Val(2));
  Val x456 = (x454 - (x455 * arg0.a[0][20]));
  Val x457 = (arg0.a[0][12] + arg0.a[0][21]);
  Val x458 = (arg0.a[0][12] * Val(2));
  Val x459 = (x457 - (x458 * arg0.a[0][21]));
  Val x460 = (arg0.a[0][2] + x366);
  Val x461 = (arg0.a[0][3] + x369);
  Val x462 = (arg0.a[0][4] + x372);
  Val x463 = (arg0.a[0][5] + x375);
  Val x464 = (arg0.a[0][6] + x378);
  Val x465 = (arg0.a[0][7] + x381);
  Val x466 = (arg0.a[0][8] + x384);
  Val x467 = (arg0.a[0][9] + x387);
  Val x468 = (arg0.a[0][10] + x390);
  Val x469 = (arg0.a[0][11] + x393);
  Val x470 = (arg0.a[0][12] + x396);
  Val x471 = (arg0.a[0][13] + x399);
  Val x472 = (arg0.a[0][14] + x402);
  Val x473 = (arg0.a[0][15] + x405);
  Val x474 = (arg0.a[0][16] + x408);
  Val x475 = (arg0.a[0][17] + x411);
  Val x476 = (arg0.a[0][18] + x414);
  Val x477 = (arg0.a[0][19] + x417);
  Val x478 = (arg0.a[0][20] + x420);
  Val x479 = (arg0.a[0][21] + x423);
  Val x480 = (arg0.a[0][22] + x426);
  Val x481 = (arg0.a[0][23] + x429);
  Val x482 = (arg0.a[0][24] + x432);
  Val x483 = (arg0.a[0][25] + x435);
  Val x484 = (arg0.a[0][26] + x438);
  Val x485 = (arg0.a[0][27] + x441);
  Val x486 = (arg0.a[0][28] + x444);
  Val x487 = (arg0.a[0][29] + x447);
  Val x488 = (arg0.a[0][30] + x450);
  Val x489 = (arg0.a[0][31] + x453);
  Val x490 = (arg0.a[0][0] + x456);
  Val x491 = (arg0.a[0][1] + x459);
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:90)
  Val x492 = (arg0.e[0][11] + arg0.e[0][25]);
  Val x493 = (arg0.e[0][11] * Val(2));
  Val x494 = (x492 - (x493 * arg0.e[0][25]));
  Val x495 = (arg0.e[0][12] + arg0.e[0][26]);
  Val x496 = (arg0.e[0][12] * Val(2));
  Val x497 = (x495 - (x496 * arg0.e[0][26]));
  Val x498 = (arg0.e[0][13] + arg0.e[0][27]);
  Val x499 = (arg0.e[0][13] * Val(2));
  Val x500 = (x498 - (x499 * arg0.e[0][27]));
  Val x501 = (arg0.e[0][14] + arg0.e[0][28]);
  Val x502 = (arg0.e[0][14] * Val(2));
  Val x503 = (x501 - (x502 * arg0.e[0][28]));
  Val x504 = (arg0.e[0][15] + arg0.e[0][29]);
  Val x505 = (arg0.e[0][15] * Val(2));
  Val x506 = (x504 - (x505 * arg0.e[0][29]));
  Val x507 = (arg0.e[0][16] + arg0.e[0][30]);
  Val x508 = (arg0.e[0][16] * Val(2));
  Val x509 = (x507 - (x508 * arg0.e[0][30]));
  Val x510 = (arg0.e[0][17] + arg0.e[0][31]);
  Val x511 = (arg0.e[0][17] * Val(2));
  Val x512 = (x510 - (x511 * arg0.e[0][31]));
  Val x513 = (arg0.e[0][18] + arg0.e[0][0]);
  Val x514 = (arg0.e[0][18] * Val(2));
  Val x515 = (x513 - (x514 * arg0.e[0][0]));
  Val x516 = (arg0.e[0][19] + arg0.e[0][1]);
  Val x517 = (arg0.e[0][19] * Val(2));
  Val x518 = (x516 - (x517 * arg0.e[0][1]));
  Val x519 = (arg0.e[0][20] + arg0.e[0][2]);
  Val x520 = (arg0.e[0][20] * Val(2));
  Val x521 = (x519 - (x520 * arg0.e[0][2]));
  Val x522 = (arg0.e[0][21] + arg0.e[0][3]);
  Val x523 = (arg0.e[0][21] * Val(2));
  Val x524 = (x522 - (x523 * arg0.e[0][3]));
  Val x525 = (arg0.e[0][22] + arg0.e[0][4]);
  Val x526 = (arg0.e[0][22] * Val(2));
  Val x527 = (x525 - (x526 * arg0.e[0][4]));
  Val x528 = (arg0.e[0][23] + arg0.e[0][5]);
  Val x529 = (arg0.e[0][23] * Val(2));
  Val x530 = (x528 - (x529 * arg0.e[0][5]));
  Val x531 = (arg0.e[0][24] + arg0.e[0][6]);
  Val x532 = (arg0.e[0][24] * Val(2));
  Val x533 = (x531 - (x532 * arg0.e[0][6]));
  Val x534 = (arg0.e[0][25] + arg0.e[0][7]);
  Val x535 = (arg0.e[0][25] * Val(2));
  Val x536 = (x534 - (x535 * arg0.e[0][7]));
  Val x537 = (arg0.e[0][26] + arg0.e[0][8]);
  Val x538 = (arg0.e[0][26] * Val(2));
  Val x539 = (x537 - (x538 * arg0.e[0][8]));
  Val x540 = (arg0.e[0][27] + arg0.e[0][9]);
  Val x541 = (arg0.e[0][27] * Val(2));
  Val x542 = (x540 - (x541 * arg0.e[0][9]));
  Val x543 = (arg0.e[0][28] + arg0.e[0][10]);
  Val x544 = (arg0.e[0][28] * Val(2));
  Val x545 = (x543 - (x544 * arg0.e[0][10]));
  Val x546 = (arg0.e[0][29] + arg0.e[0][11]);
  Val x547 = (arg0.e[0][29] * Val(2));
  Val x548 = (x546 - (x547 * arg0.e[0][11]));
  Val x549 = (arg0.e[0][30] + arg0.e[0][12]);
  Val x550 = (arg0.e[0][30] * Val(2));
  Val x551 = (x549 - (x550 * arg0.e[0][12]));
  Val x552 = (arg0.e[0][31] + arg0.e[0][13]);
  Val x553 = (arg0.e[0][31] * Val(2));
  Val x554 = (x552 - (x553 * arg0.e[0][13]));
  Val x555 = (arg0.e[0][0] + arg0.e[0][14]);
  Val x556 = (arg0.e[0][0] * Val(2));
  Val x557 = (x555 - (x556 * arg0.e[0][14]));
  Val x558 = (arg0.e[0][1] + arg0.e[0][15]);
  Val x559 = (arg0.e[0][1] * Val(2));
  Val x560 = (x558 - (x559 * arg0.e[0][15]));
  Val x561 = (arg0.e[0][2] + arg0.e[0][16]);
  Val x562 = (arg0.e[0][2] * Val(2));
  Val x563 = (x561 - (x562 * arg0.e[0][16]));
  Val x564 = (arg0.e[0][3] + arg0.e[0][17]);
  Val x565 = (arg0.e[0][3] * Val(2));
  Val x566 = (x564 - (x565 * arg0.e[0][17]));
  Val x567 = (arg0.e[0][4] + arg0.e[0][18]);
  Val x568 = (arg0.e[0][4] * Val(2));
  Val x569 = (x567 - (x568 * arg0.e[0][18]));
  Val x570 = (arg0.e[0][5] + arg0.e[0][19]);
  Val x571 = (arg0.e[0][5] * Val(2));
  Val x572 = (x570 - (x571 * arg0.e[0][19]));
  Val x573 = (arg0.e[0][6] + arg0.e[0][20]);
  Val x574 = (arg0.e[0][6] * Val(2));
  Val x575 = (x573 - (x574 * arg0.e[0][20]));
  Val x576 = (arg0.e[0][7] + arg0.e[0][21]);
  Val x577 = (arg0.e[0][7] * Val(2));
  Val x578 = (x576 - (x577 * arg0.e[0][21]));
  Val x579 = (arg0.e[0][8] + arg0.e[0][22]);
  Val x580 = (arg0.e[0][8] * Val(2));
  Val x581 = (x579 - (x580 * arg0.e[0][22]));
  Val x582 = (arg0.e[0][9] + arg0.e[0][23]);
  Val x583 = (arg0.e[0][9] * Val(2));
  Val x584 = (x582 - (x583 * arg0.e[0][23]));
  Val x585 = (arg0.e[0][10] + arg0.e[0][24]);
  Val x586 = (arg0.e[0][10] * Val(2));
  Val x587 = (x585 - (x586 * arg0.e[0][24]));
  Val x588 = (arg0.e[0][6] + x494);
  Val x589 = (arg0.e[0][7] + x497);
  Val x590 = (arg0.e[0][8] + x500);
  Val x591 = (arg0.e[0][9] + x503);
  Val x592 = (arg0.e[0][10] + x506);
  Val x593 = (arg0.e[0][11] + x509);
  Val x594 = (arg0.e[0][12] + x512);
  Val x595 = (arg0.e[0][13] + x515);
  Val x596 = (arg0.e[0][14] + x518);
  Val x597 = (arg0.e[0][15] + x521);
  Val x598 = (arg0.e[0][16] + x524);
  Val x599 = (arg0.e[0][17] + x527);
  Val x600 = (arg0.e[0][18] + x530);
  Val x601 = (arg0.e[0][19] + x533);
  Val x602 = (arg0.e[0][20] + x536);
  Val x603 = (arg0.e[0][21] + x539);
  Val x604 = (arg0.e[0][22] + x542);
  Val x605 = (arg0.e[0][23] + x545);
  Val x606 = (arg0.e[0][24] + x548);
  Val x607 = (arg0.e[0][25] + x551);
  Val x608 = (arg0.e[0][26] + x554);
  Val x609 = (arg0.e[0][27] + x557);
  Val x610 = (arg0.e[0][28] + x560);
  Val x611 = (arg0.e[0][29] + x563);
  Val x612 = (arg0.e[0][30] + x566);
  Val x613 = (arg0.e[0][31] + x569);
  Val x614 = (arg0.e[0][0] + x572);
  Val x615 = (arg0.e[0][1] + x575);
  Val x616 = (arg0.e[0][2] + x578);
  Val x617 = (arg0.e[0][3] + x581);
  Val x618 = (arg0.e[0][4] + x584);
  Val x619 = (arg0.e[0][5] + x587);
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:72)
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:91)
  Val x620 = (x360[1]._super._super * Val(2));
  Val x621 = (x360[2]._super._super * Val(4));
  Val x622 = (x360[3]._super._super * Val(8));
  Val x623 = (x360[4]._super._super * Val(16));
  Val x624 = (x360[5]._super._super * Val(32));
  Val x625 = (x360[6]._super._super * Val(64));
  Val x626 = (x360[7]._super._super * Val(128));
  Val x627 = (x360[8]._super._super * Val(256));
  Val x628 = (x360[9]._super._super * Val(512));
  Val x629 = (x360[10]._super._super * Val(1024));
  Val x630 = (x360[11]._super._super * Val(2048));
  Val x631 = (x360[12]._super._super * Val(4096));
  Val x632 = (x360[13]._super._super * Val(8192));
  Val x633 = (x360[14]._super._super * Val(16384));
  Val x634 = (x360[15]._super._super * Val(32768));
  Val x635 = (x360[0]._super._super + x620);
  Val x636 = (((x635 + x621) + x622) + x623);
  Val x637 = (((x636 + x624) + x625) + x626);
  Val x638 = (((x637 + x627) + x628) + x629);
  Val x639 = (((x638 + x630) + x631) + x632);
  Val x640 = (x360[17]._super._super * Val(2));
  Val x641 = (x360[18]._super._super * Val(4));
  Val x642 = (x360[19]._super._super * Val(8));
  Val x643 = (x360[20]._super._super * Val(16));
  Val x644 = (x360[21]._super._super * Val(32));
  Val x645 = (x360[22]._super._super * Val(64));
  Val x646 = (x360[23]._super._super * Val(128));
  Val x647 = (x360[24]._super._super * Val(256));
  Val x648 = (x360[25]._super._super * Val(512));
  Val x649 = (x360[26]._super._super * Val(1024));
  Val x650 = (x360[27]._super._super * Val(2048));
  Val x651 = (x360[28]._super._super * Val(4096));
  Val x652 = (x360[29]._super._super * Val(8192));
  Val x653 = (x360[30]._super._super * Val(16384));
  Val x654 = (x360[31]._super._super * Val(32768));
  Val x655 = (x360[16]._super._super + x640);
  Val x656 = (((x655 + x641) + x642) + x643);
  Val x657 = (((x656 + x644) + x645) + x646);
  Val x658 = (((x657 + x647) + x648) + x649);
  Val x659 = (((x658 + x650) + x651) + x652);
  Val x660 = (arg0.e[3][1] * Val(2));
  Val x661 = (arg0.e[3][2] * Val(4));
  Val x662 = (arg0.e[3][3] * Val(8));
  Val x663 = (arg0.e[3][4] * Val(16));
  Val x664 = (arg0.e[3][5] * Val(32));
  Val x665 = (arg0.e[3][6] * Val(64));
  Val x666 = (arg0.e[3][7] * Val(128));
  Val x667 = (arg0.e[3][8] * Val(256));
  Val x668 = (arg0.e[3][9] * Val(512));
  Val x669 = (arg0.e[3][10] * Val(1024));
  Val x670 = (arg0.e[3][11] * Val(2048));
  Val x671 = (arg0.e[3][12] * Val(4096));
  Val x672 = (arg0.e[3][13] * Val(8192));
  Val x673 = (arg0.e[3][14] * Val(16384));
  Val x674 = (arg0.e[3][15] * Val(32768));
  Val x675 = (arg0.e[3][0] + x660);
  Val x676 = (((x675 + x661) + x662) + x663);
  Val x677 = (((x676 + x664) + x665) + x666);
  Val x678 = (((x677 + x667) + x668) + x669);
  Val x679 = (((x678 + x670) + x671) + x672);
  Val x680 = (arg0.e[3][17] * Val(2));
  Val x681 = (arg0.e[3][18] * Val(4));
  Val x682 = (arg0.e[3][19] * Val(8));
  Val x683 = (arg0.e[3][20] * Val(16));
  Val x684 = (arg0.e[3][21] * Val(32));
  Val x685 = (arg0.e[3][22] * Val(64));
  Val x686 = (arg0.e[3][23] * Val(128));
  Val x687 = (arg0.e[3][24] * Val(256));
  Val x688 = (arg0.e[3][25] * Val(512));
  Val x689 = (arg0.e[3][26] * Val(1024));
  Val x690 = (arg0.e[3][27] * Val(2048));
  Val x691 = (arg0.e[3][28] * Val(4096));
  Val x692 = (arg0.e[3][29] * Val(8192));
  Val x693 = (arg0.e[3][30] * Val(16384));
  Val x694 = (arg0.e[3][31] * Val(32768));
  Val x695 = (arg0.e[3][16] + x680);
  Val x696 = (((x695 + x681) + x682) + x683);
  Val x697 = (((x696 + x684) + x685) + x686);
  Val x698 = (((x697 + x687) + x688) + x689);
  Val x699 = (((x698 + x690) + x691) + x692);
  // ChU32(zirgen/circuit/keccak2/sha2.zir:25)
  Val x700 = (arg0.e[0][0] * arg0.e[1][0]);
  Val x701 = (Val(1) - arg0.e[0][0]);
  Val x702 = (arg0.e[0][1] * arg0.e[1][1]);
  Val x703 = (Val(1) - arg0.e[0][1]);
  Val x704 = (arg0.e[0][2] * arg0.e[1][2]);
  Val x705 = (Val(1) - arg0.e[0][2]);
  Val x706 = (arg0.e[0][3] * arg0.e[1][3]);
  Val x707 = (Val(1) - arg0.e[0][3]);
  Val x708 = (arg0.e[0][4] * arg0.e[1][4]);
  Val x709 = (Val(1) - arg0.e[0][4]);
  Val x710 = (arg0.e[0][5] * arg0.e[1][5]);
  Val x711 = (Val(1) - arg0.e[0][5]);
  Val x712 = (arg0.e[0][6] * arg0.e[1][6]);
  Val x713 = (Val(1) - arg0.e[0][6]);
  Val x714 = (arg0.e[0][7] * arg0.e[1][7]);
  Val x715 = (Val(1) - arg0.e[0][7]);
  Val x716 = (arg0.e[0][8] * arg0.e[1][8]);
  Val x717 = (Val(1) - arg0.e[0][8]);
  Val x718 = (arg0.e[0][9] * arg0.e[1][9]);
  Val x719 = (Val(1) - arg0.e[0][9]);
  Val x720 = (arg0.e[0][10] * arg0.e[1][10]);
  Val x721 = (Val(1) - arg0.e[0][10]);
  Val x722 = (arg0.e[0][11] * arg0.e[1][11]);
  Val x723 = (Val(1) - arg0.e[0][11]);
  Val x724 = (arg0.e[0][12] * arg0.e[1][12]);
  Val x725 = (Val(1) - arg0.e[0][12]);
  Val x726 = (arg0.e[0][13] * arg0.e[1][13]);
  Val x727 = (Val(1) - arg0.e[0][13]);
  Val x728 = (arg0.e[0][14] * arg0.e[1][14]);
  Val x729 = (Val(1) - arg0.e[0][14]);
  Val x730 = (arg0.e[0][15] * arg0.e[1][15]);
  Val x731 = (Val(1) - arg0.e[0][15]);
  Val x732 = (arg0.e[0][16] * arg0.e[1][16]);
  Val x733 = (Val(1) - arg0.e[0][16]);
  Val x734 = (arg0.e[0][17] * arg0.e[1][17]);
  Val x735 = (Val(1) - arg0.e[0][17]);
  Val x736 = (arg0.e[0][18] * arg0.e[1][18]);
  Val x737 = (Val(1) - arg0.e[0][18]);
  Val x738 = (arg0.e[0][19] * arg0.e[1][19]);
  Val x739 = (Val(1) - arg0.e[0][19]);
  Val x740 = (arg0.e[0][20] * arg0.e[1][20]);
  Val x741 = (Val(1) - arg0.e[0][20]);
  Val x742 = (arg0.e[0][21] * arg0.e[1][21]);
  Val x743 = (Val(1) - arg0.e[0][21]);
  Val x744 = (arg0.e[0][22] * arg0.e[1][22]);
  Val x745 = (Val(1) - arg0.e[0][22]);
  Val x746 = (arg0.e[0][23] * arg0.e[1][23]);
  Val x747 = (Val(1) - arg0.e[0][23]);
  Val x748 = (arg0.e[0][24] * arg0.e[1][24]);
  Val x749 = (Val(1) - arg0.e[0][24]);
  Val x750 = (arg0.e[0][25] * arg0.e[1][25]);
  Val x751 = (Val(1) - arg0.e[0][25]);
  Val x752 = (arg0.e[0][26] * arg0.e[1][26]);
  Val x753 = (Val(1) - arg0.e[0][26]);
  Val x754 = (arg0.e[0][27] * arg0.e[1][27]);
  Val x755 = (Val(1) - arg0.e[0][27]);
  Val x756 = (arg0.e[0][28] * arg0.e[1][28]);
  Val x757 = (Val(1) - arg0.e[0][28]);
  Val x758 = (arg0.e[0][29] * arg0.e[1][29]);
  Val x759 = (Val(1) - arg0.e[0][29]);
  Val x760 = (arg0.e[0][30] * arg0.e[1][30]);
  Val x761 = (Val(1) - arg0.e[0][30]);
  Val x762 = (arg0.e[0][31] * arg0.e[1][31]);
  Val x763 = (Val(1) - arg0.e[0][31]);
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:72)
  Val x764 = ((x700 + (x701 * arg0.e[2][0])) + ((x702 + (x703 * arg0.e[2][1])) * Val(2)));
  Val x765 = ((x764 + ((x704 + (x705 * arg0.e[2][2])) * Val(4))) +
              ((x706 + (x707 * arg0.e[2][3])) * Val(8)));
  Val x766 = ((x765 + ((x708 + (x709 * arg0.e[2][4])) * Val(16))) +
              ((x710 + (x711 * arg0.e[2][5])) * Val(32)));
  Val x767 = ((x766 + ((x712 + (x713 * arg0.e[2][6])) * Val(64))) +
              ((x714 + (x715 * arg0.e[2][7])) * Val(128)));
  Val x768 = ((x767 + ((x716 + (x717 * arg0.e[2][8])) * Val(256))) +
              ((x718 + (x719 * arg0.e[2][9])) * Val(512)));
  Val x769 = ((x768 + ((x720 + (x721 * arg0.e[2][10])) * Val(1024))) +
              ((x722 + (x723 * arg0.e[2][11])) * Val(2048)));
  Val x770 = ((x769 + ((x724 + (x725 * arg0.e[2][12])) * Val(4096))) +
              ((x726 + (x727 * arg0.e[2][13])) * Val(8192)));
  Val x771 = ((x770 + ((x728 + (x729 * arg0.e[2][14])) * Val(16384))) +
              ((x730 + (x731 * arg0.e[2][15])) * Val(32768)));
  Val x772 = ((x732 + (x733 * arg0.e[2][16])) + ((x734 + (x735 * arg0.e[2][17])) * Val(2)));
  Val x773 = ((x772 + ((x736 + (x737 * arg0.e[2][18])) * Val(4))) +
              ((x738 + (x739 * arg0.e[2][19])) * Val(8)));
  Val x774 = ((x773 + ((x740 + (x741 * arg0.e[2][20])) * Val(16))) +
              ((x742 + (x743 * arg0.e[2][21])) * Val(32)));
  Val x775 = ((x774 + ((x744 + (x745 * arg0.e[2][22])) * Val(64))) +
              ((x746 + (x747 * arg0.e[2][23])) * Val(128)));
  Val x776 = ((x775 + ((x748 + (x749 * arg0.e[2][24])) * Val(256))) +
              ((x750 + (x751 * arg0.e[2][25])) * Val(512)));
  Val x777 = ((x776 + ((x752 + (x753 * arg0.e[2][26])) * Val(1024))) +
              ((x754 + (x755 * arg0.e[2][27])) * Val(2048)));
  Val x778 = ((x777 + ((x756 + (x757 * arg0.e[2][28])) * Val(4096))) +
              ((x758 + (x759 * arg0.e[2][29])) * Val(8192)));
  Val x779 = ((x778 + ((x760 + (x761 * arg0.e[2][30])) * Val(16384))) +
              ((x762 + (x763 * arg0.e[2][31])) * Val(32768)));
  Val x780 = ((x588 - (x574 * x494)) + ((x589 - (x577 * x497)) * Val(2)));
  Val x781 = ((x780 + ((x590 - (x580 * x500)) * Val(4))) + ((x591 - (x583 * x503)) * Val(8)));
  Val x782 = ((x781 + ((x592 - (x586 * x506)) * Val(16))) + ((x593 - (x493 * x509)) * Val(32)));
  Val x783 = ((x782 + ((x594 - (x496 * x512)) * Val(64))) + ((x595 - (x499 * x515)) * Val(128)));
  Val x784 = ((x783 + ((x596 - (x502 * x518)) * Val(256))) + ((x597 - (x505 * x521)) * Val(512)));
  Val x785 = ((x784 + ((x598 - (x508 * x524)) * Val(1024))) + ((x599 - (x511 * x527)) * Val(2048)));
  Val x786 = ((x785 + ((x600 - (x514 * x530)) * Val(4096))) + ((x601 - (x517 * x533)) * Val(8192)));
  Val x787 =
      ((x786 + ((x602 - (x520 * x536)) * Val(16384))) + ((x603 - (x523 * x539)) * Val(32768)));
  Val x788 = ((x604 - (x526 * x542)) + ((x605 - (x529 * x545)) * Val(2)));
  Val x789 = ((x788 + ((x606 - (x532 * x548)) * Val(4))) + ((x607 - (x535 * x551)) * Val(8)));
  Val x790 = ((x789 + ((x608 - (x538 * x554)) * Val(16))) + ((x609 - (x541 * x557)) * Val(32)));
  Val x791 = ((x790 + ((x610 - (x544 * x560)) * Val(64))) + ((x611 - (x547 * x563)) * Val(128)));
  Val x792 = ((x791 + ((x612 - (x550 * x566)) * Val(256))) + ((x613 - (x553 * x569)) * Val(512)));
  Val x793 = ((x792 + ((x614 - (x556 * x572)) * Val(1024))) + ((x615 - (x559 * x575)) * Val(2048)));
  Val x794 = ((x793 + ((x616 - (x562 * x578)) * Val(4096))) + ((x617 - (x565 * x581)) * Val(8192)));
  Val x795 =
      ((x794 + ((x618 - (x568 * x584)) * Val(16384))) + ((x619 - (x571 * x587)) * Val(32768)));
  // Add2(zirgen/circuit/keccak2/sha2.zir:30)
  Val x796 = (((x679 + x673) + x674) + (x771 + x787));
  Val x797 = (((x699 + x693) + x694) + (x779 + x795));
  Val x798 = (((x639 + x633) + x634) + (arg1[0] + x796));
  Val x799 = (((x659 + x653) + x654) + (arg1[1] + x797));
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:92)
  Val x800 = (arg0.a[0][0] * arg0.a[1][0]);
  Val x801 = (Val(1) - arg0.a[2][0]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x802 = (Val(1) - arg0.a[1][0]);
  Val x803 = (arg0.a[0][0] * x802);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x804 = (Val(1) - arg0.a[0][0]);
  Val x805 = ((x804 * arg0.a[1][0]) * arg0.a[2][0]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x806 = (((x800 * x801) + (x803 * arg0.a[2][0])) + x805);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x807 = (arg0.a[0][1] * arg0.a[1][1]);
  Val x808 = (Val(1) - arg0.a[2][1]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x809 = (Val(1) - arg0.a[1][1]);
  Val x810 = (arg0.a[0][1] * x809);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x811 = (Val(1) - arg0.a[0][1]);
  Val x812 = ((x811 * arg0.a[1][1]) * arg0.a[2][1]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x813 = (((x807 * x808) + (x810 * arg0.a[2][1])) + x812);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x814 = (arg0.a[0][2] * arg0.a[1][2]);
  Val x815 = (Val(1) - arg0.a[2][2]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x816 = (Val(1) - arg0.a[1][2]);
  Val x817 = (arg0.a[0][2] * x816);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x818 = (Val(1) - arg0.a[0][2]);
  Val x819 = ((x818 * arg0.a[1][2]) * arg0.a[2][2]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x820 = (((x814 * x815) + (x817 * arg0.a[2][2])) + x819);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x821 = (arg0.a[0][3] * arg0.a[1][3]);
  Val x822 = (Val(1) - arg0.a[2][3]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x823 = (Val(1) - arg0.a[1][3]);
  Val x824 = (arg0.a[0][3] * x823);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x825 = (Val(1) - arg0.a[0][3]);
  Val x826 = ((x825 * arg0.a[1][3]) * arg0.a[2][3]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x827 = (((x821 * x822) + (x824 * arg0.a[2][3])) + x826);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x828 = (arg0.a[0][4] * arg0.a[1][4]);
  Val x829 = (Val(1) - arg0.a[2][4]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x830 = (Val(1) - arg0.a[1][4]);
  Val x831 = (arg0.a[0][4] * x830);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x832 = (Val(1) - arg0.a[0][4]);
  Val x833 = ((x832 * arg0.a[1][4]) * arg0.a[2][4]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x834 = (((x828 * x829) + (x831 * arg0.a[2][4])) + x833);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x835 = (arg0.a[0][5] * arg0.a[1][5]);
  Val x836 = (Val(1) - arg0.a[2][5]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x837 = (Val(1) - arg0.a[1][5]);
  Val x838 = (arg0.a[0][5] * x837);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x839 = (Val(1) - arg0.a[0][5]);
  Val x840 = ((x839 * arg0.a[1][5]) * arg0.a[2][5]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x841 = (((x835 * x836) + (x838 * arg0.a[2][5])) + x840);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x842 = (arg0.a[0][6] * arg0.a[1][6]);
  Val x843 = (Val(1) - arg0.a[2][6]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x844 = (Val(1) - arg0.a[1][6]);
  Val x845 = (arg0.a[0][6] * x844);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x846 = (Val(1) - arg0.a[0][6]);
  Val x847 = ((x846 * arg0.a[1][6]) * arg0.a[2][6]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x848 = (((x842 * x843) + (x845 * arg0.a[2][6])) + x847);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x849 = (arg0.a[0][7] * arg0.a[1][7]);
  Val x850 = (Val(1) - arg0.a[2][7]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x851 = (Val(1) - arg0.a[1][7]);
  Val x852 = (arg0.a[0][7] * x851);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x853 = (Val(1) - arg0.a[0][7]);
  Val x854 = ((x853 * arg0.a[1][7]) * arg0.a[2][7]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x855 = (((x849 * x850) + (x852 * arg0.a[2][7])) + x854);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x856 = (arg0.a[0][8] * arg0.a[1][8]);
  Val x857 = (Val(1) - arg0.a[2][8]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x858 = (Val(1) - arg0.a[1][8]);
  Val x859 = (arg0.a[0][8] * x858);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x860 = (Val(1) - arg0.a[0][8]);
  Val x861 = ((x860 * arg0.a[1][8]) * arg0.a[2][8]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x862 = (((x856 * x857) + (x859 * arg0.a[2][8])) + x861);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x863 = (arg0.a[0][9] * arg0.a[1][9]);
  Val x864 = (Val(1) - arg0.a[2][9]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x865 = (Val(1) - arg0.a[1][9]);
  Val x866 = (arg0.a[0][9] * x865);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x867 = (Val(1) - arg0.a[0][9]);
  Val x868 = ((x867 * arg0.a[1][9]) * arg0.a[2][9]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x869 = (((x863 * x864) + (x866 * arg0.a[2][9])) + x868);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x870 = (arg0.a[0][10] * arg0.a[1][10]);
  Val x871 = (Val(1) - arg0.a[2][10]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x872 = (Val(1) - arg0.a[1][10]);
  Val x873 = (arg0.a[0][10] * x872);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x874 = (Val(1) - arg0.a[0][10]);
  Val x875 = ((x874 * arg0.a[1][10]) * arg0.a[2][10]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x876 = (((x870 * x871) + (x873 * arg0.a[2][10])) + x875);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x877 = (arg0.a[0][11] * arg0.a[1][11]);
  Val x878 = (Val(1) - arg0.a[2][11]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x879 = (Val(1) - arg0.a[1][11]);
  Val x880 = (arg0.a[0][11] * x879);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x881 = (Val(1) - arg0.a[0][11]);
  Val x882 = ((x881 * arg0.a[1][11]) * arg0.a[2][11]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x883 = (((x877 * x878) + (x880 * arg0.a[2][11])) + x882);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x884 = (arg0.a[0][12] * arg0.a[1][12]);
  Val x885 = (Val(1) - arg0.a[2][12]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x886 = (Val(1) - arg0.a[1][12]);
  Val x887 = (arg0.a[0][12] * x886);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x888 = (Val(1) - arg0.a[0][12]);
  Val x889 = ((x888 * arg0.a[1][12]) * arg0.a[2][12]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x890 = (((x884 * x885) + (x887 * arg0.a[2][12])) + x889);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x891 = (arg0.a[0][13] * arg0.a[1][13]);
  Val x892 = (Val(1) - arg0.a[2][13]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x893 = (Val(1) - arg0.a[1][13]);
  Val x894 = (arg0.a[0][13] * x893);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x895 = (Val(1) - arg0.a[0][13]);
  Val x896 = ((x895 * arg0.a[1][13]) * arg0.a[2][13]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x897 = (((x891 * x892) + (x894 * arg0.a[2][13])) + x896);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x898 = (arg0.a[0][14] * arg0.a[1][14]);
  Val x899 = (Val(1) - arg0.a[2][14]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x900 = (Val(1) - arg0.a[1][14]);
  Val x901 = (arg0.a[0][14] * x900);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x902 = (Val(1) - arg0.a[0][14]);
  Val x903 = ((x902 * arg0.a[1][14]) * arg0.a[2][14]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x904 = (((x898 * x899) + (x901 * arg0.a[2][14])) + x903);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x905 = (arg0.a[0][15] * arg0.a[1][15]);
  Val x906 = (Val(1) - arg0.a[2][15]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x907 = (Val(1) - arg0.a[1][15]);
  Val x908 = (arg0.a[0][15] * x907);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x909 = (Val(1) - arg0.a[0][15]);
  Val x910 = ((x909 * arg0.a[1][15]) * arg0.a[2][15]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x911 = (((x905 * x906) + (x908 * arg0.a[2][15])) + x910);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x912 = (arg0.a[0][16] * arg0.a[1][16]);
  Val x913 = (Val(1) - arg0.a[2][16]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x914 = (Val(1) - arg0.a[1][16]);
  Val x915 = (arg0.a[0][16] * x914);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x916 = (Val(1) - arg0.a[0][16]);
  Val x917 = ((x916 * arg0.a[1][16]) * arg0.a[2][16]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x918 = (((x912 * x913) + (x915 * arg0.a[2][16])) + x917);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x919 = (arg0.a[0][17] * arg0.a[1][17]);
  Val x920 = (Val(1) - arg0.a[2][17]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x921 = (Val(1) - arg0.a[1][17]);
  Val x922 = (arg0.a[0][17] * x921);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x923 = (Val(1) - arg0.a[0][17]);
  Val x924 = ((x923 * arg0.a[1][17]) * arg0.a[2][17]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x925 = (((x919 * x920) + (x922 * arg0.a[2][17])) + x924);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x926 = (arg0.a[0][18] * arg0.a[1][18]);
  Val x927 = (Val(1) - arg0.a[2][18]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x928 = (Val(1) - arg0.a[1][18]);
  Val x929 = (arg0.a[0][18] * x928);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x930 = (Val(1) - arg0.a[0][18]);
  Val x931 = ((x930 * arg0.a[1][18]) * arg0.a[2][18]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x932 = (((x926 * x927) + (x929 * arg0.a[2][18])) + x931);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x933 = (arg0.a[0][19] * arg0.a[1][19]);
  Val x934 = (Val(1) - arg0.a[2][19]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x935 = (Val(1) - arg0.a[1][19]);
  Val x936 = (arg0.a[0][19] * x935);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x937 = (Val(1) - arg0.a[0][19]);
  Val x938 = ((x937 * arg0.a[1][19]) * arg0.a[2][19]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x939 = (((x933 * x934) + (x936 * arg0.a[2][19])) + x938);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x940 = (arg0.a[0][20] * arg0.a[1][20]);
  Val x941 = (Val(1) - arg0.a[2][20]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x942 = (Val(1) - arg0.a[1][20]);
  Val x943 = (arg0.a[0][20] * x942);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x944 = (Val(1) - arg0.a[0][20]);
  Val x945 = ((x944 * arg0.a[1][20]) * arg0.a[2][20]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x946 = (((x940 * x941) + (x943 * arg0.a[2][20])) + x945);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x947 = (arg0.a[0][21] * arg0.a[1][21]);
  Val x948 = (Val(1) - arg0.a[2][21]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x949 = (Val(1) - arg0.a[1][21]);
  Val x950 = (arg0.a[0][21] * x949);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x951 = (Val(1) - arg0.a[0][21]);
  Val x952 = ((x951 * arg0.a[1][21]) * arg0.a[2][21]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x953 = (((x947 * x948) + (x950 * arg0.a[2][21])) + x952);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x954 = (arg0.a[0][22] * arg0.a[1][22]);
  Val x955 = (Val(1) - arg0.a[2][22]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x956 = (Val(1) - arg0.a[1][22]);
  Val x957 = (arg0.a[0][22] * x956);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x958 = (Val(1) - arg0.a[0][22]);
  Val x959 = ((x958 * arg0.a[1][22]) * arg0.a[2][22]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x960 = (((x954 * x955) + (x957 * arg0.a[2][22])) + x959);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x961 = (arg0.a[0][23] * arg0.a[1][23]);
  Val x962 = (Val(1) - arg0.a[2][23]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x963 = (Val(1) - arg0.a[1][23]);
  Val x964 = (arg0.a[0][23] * x963);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x965 = (Val(1) - arg0.a[0][23]);
  Val x966 = ((x965 * arg0.a[1][23]) * arg0.a[2][23]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x967 = (((x961 * x962) + (x964 * arg0.a[2][23])) + x966);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x968 = (arg0.a[0][24] * arg0.a[1][24]);
  Val x969 = (Val(1) - arg0.a[2][24]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x970 = (Val(1) - arg0.a[1][24]);
  Val x971 = (arg0.a[0][24] * x970);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x972 = (Val(1) - arg0.a[0][24]);
  Val x973 = ((x972 * arg0.a[1][24]) * arg0.a[2][24]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x974 = (((x968 * x969) + (x971 * arg0.a[2][24])) + x973);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x975 = (arg0.a[0][25] * arg0.a[1][25]);
  Val x976 = (Val(1) - arg0.a[2][25]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x977 = (Val(1) - arg0.a[1][25]);
  Val x978 = (arg0.a[0][25] * x977);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x979 = (Val(1) - arg0.a[0][25]);
  Val x980 = ((x979 * arg0.a[1][25]) * arg0.a[2][25]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x981 = (((x975 * x976) + (x978 * arg0.a[2][25])) + x980);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x982 = (arg0.a[0][26] * arg0.a[1][26]);
  Val x983 = (Val(1) - arg0.a[2][26]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x984 = (Val(1) - arg0.a[1][26]);
  Val x985 = (arg0.a[0][26] * x984);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x986 = (Val(1) - arg0.a[0][26]);
  Val x987 = ((x986 * arg0.a[1][26]) * arg0.a[2][26]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x988 = (((x982 * x983) + (x985 * arg0.a[2][26])) + x987);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x989 = (arg0.a[0][27] * arg0.a[1][27]);
  Val x990 = (Val(1) - arg0.a[2][27]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x991 = (Val(1) - arg0.a[1][27]);
  Val x992 = (arg0.a[0][27] * x991);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x993 = (Val(1) - arg0.a[0][27]);
  Val x994 = ((x993 * arg0.a[1][27]) * arg0.a[2][27]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x995 = (((x989 * x990) + (x992 * arg0.a[2][27])) + x994);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x996 = (arg0.a[0][28] * arg0.a[1][28]);
  Val x997 = (Val(1) - arg0.a[2][28]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x998 = (Val(1) - arg0.a[1][28]);
  Val x999 = (arg0.a[0][28] * x998);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x1000 = (Val(1) - arg0.a[0][28]);
  Val x1001 = ((x1000 * arg0.a[1][28]) * arg0.a[2][28]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1002 = (((x996 * x997) + (x999 * arg0.a[2][28])) + x1001);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x1003 = (arg0.a[0][29] * arg0.a[1][29]);
  Val x1004 = (Val(1) - arg0.a[2][29]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1005 = (Val(1) - arg0.a[1][29]);
  Val x1006 = (arg0.a[0][29] * x1005);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x1007 = (Val(1) - arg0.a[0][29]);
  Val x1008 = ((x1007 * arg0.a[1][29]) * arg0.a[2][29]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1009 = (((x1003 * x1004) + (x1006 * arg0.a[2][29])) + x1008);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x1010 = (arg0.a[0][30] * arg0.a[1][30]);
  Val x1011 = (Val(1) - arg0.a[2][30]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1012 = (Val(1) - arg0.a[1][30]);
  Val x1013 = (arg0.a[0][30] * x1012);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x1014 = (Val(1) - arg0.a[0][30]);
  Val x1015 = ((x1014 * arg0.a[1][30]) * arg0.a[2][30]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1016 = (((x1010 * x1011) + (x1013 * arg0.a[2][30])) + x1015);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:16)
  Val x1017 = (arg0.a[0][31] * arg0.a[1][31]);
  Val x1018 = (Val(1) - arg0.a[2][31]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1019 = (Val(1) - arg0.a[1][31]);
  Val x1020 = (arg0.a[0][31] * x1019);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:18)
  Val x1021 = (Val(1) - arg0.a[0][31]);
  Val x1022 = ((x1021 * arg0.a[1][31]) * arg0.a[2][31]);
  // MajU32(zirgen/circuit/keccak2/sha2.zir:17)
  Val x1023 = (((x1017 * x1018) + (x1020 * arg0.a[2][31])) + x1022);
  // Pack(zirgen/circuit/keccak2/pack.zir:32)
  // Pack32(zirgen/circuit/keccak2/sha2.zir:72)
  Val x1024 = ((x806 + (x800 * arg0.a[2][0])) + ((x813 + (x807 * arg0.a[2][1])) * Val(2)));
  Val x1025 = ((x1024 + ((x820 + (x814 * arg0.a[2][2])) * Val(4))) +
               ((x827 + (x821 * arg0.a[2][3])) * Val(8)));
  Val x1026 = ((x1025 + ((x834 + (x828 * arg0.a[2][4])) * Val(16))) +
               ((x841 + (x835 * arg0.a[2][5])) * Val(32)));
  Val x1027 = ((x1026 + ((x848 + (x842 * arg0.a[2][6])) * Val(64))) +
               ((x855 + (x849 * arg0.a[2][7])) * Val(128)));
  Val x1028 = ((x1027 + ((x862 + (x856 * arg0.a[2][8])) * Val(256))) +
               ((x869 + (x863 * arg0.a[2][9])) * Val(512)));
  Val x1029 = ((x1028 + ((x876 + (x870 * arg0.a[2][10])) * Val(1024))) +
               ((x883 + (x877 * arg0.a[2][11])) * Val(2048)));
  Val x1030 = ((x1029 + ((x890 + (x884 * arg0.a[2][12])) * Val(4096))) +
               ((x897 + (x891 * arg0.a[2][13])) * Val(8192)));
  Val x1031 = ((x1030 + ((x904 + (x898 * arg0.a[2][14])) * Val(16384))) +
               ((x911 + (x905 * arg0.a[2][15])) * Val(32768)));
  Val x1032 = ((x918 + (x912 * arg0.a[2][16])) + ((x925 + (x919 * arg0.a[2][17])) * Val(2)));
  Val x1033 = ((x1032 + ((x932 + (x926 * arg0.a[2][18])) * Val(4))) +
               ((x939 + (x933 * arg0.a[2][19])) * Val(8)));
  Val x1034 = ((x1033 + ((x946 + (x940 * arg0.a[2][20])) * Val(16))) +
               ((x953 + (x947 * arg0.a[2][21])) * Val(32)));
  Val x1035 = ((x1034 + ((x960 + (x954 * arg0.a[2][22])) * Val(64))) +
               ((x967 + (x961 * arg0.a[2][23])) * Val(128)));
  Val x1036 = ((x1035 + ((x974 + (x968 * arg0.a[2][24])) * Val(256))) +
               ((x981 + (x975 * arg0.a[2][25])) * Val(512)));
  Val x1037 = ((x1036 + ((x988 + (x982 * arg0.a[2][26])) * Val(1024))) +
               ((x995 + (x989 * arg0.a[2][27])) * Val(2048)));
  Val x1038 = ((x1037 + ((x1002 + (x996 * arg0.a[2][28])) * Val(4096))) +
               ((x1009 + (x1003 * arg0.a[2][29])) * Val(8192)));
  Val x1039 = ((x1038 + ((x1016 + (x1010 * arg0.a[2][30])) * Val(16384))) +
               ((x1023 + (x1017 * arg0.a[2][31])) * Val(32768)));
  Val x1040 = ((x460 - (x428 * x366)) + ((x461 - (x431 * x369)) * Val(2)));
  Val x1041 = ((x1040 + ((x462 - (x434 * x372)) * Val(4))) + ((x463 - (x437 * x375)) * Val(8)));
  Val x1042 = ((x1041 + ((x464 - (x440 * x378)) * Val(16))) + ((x465 - (x443 * x381)) * Val(32)));
  Val x1043 = ((x1042 + ((x466 - (x446 * x384)) * Val(64))) + ((x467 - (x449 * x387)) * Val(128)));
  Val x1044 = ((x1043 + ((x468 - (x452 * x390)) * Val(256))) + ((x469 - (x455 * x393)) * Val(512)));
  Val x1045 =
      ((x1044 + ((x470 - (x458 * x396)) * Val(1024))) + ((x471 - (x365 * x399)) * Val(2048)));
  Val x1046 =
      ((x1045 + ((x472 - (x368 * x402)) * Val(4096))) + ((x473 - (x371 * x405)) * Val(8192)));
  Val x1047 =
      ((x1046 + ((x474 - (x374 * x408)) * Val(16384))) + ((x475 - (x377 * x411)) * Val(32768)));
  Val x1048 = ((x476 - (x380 * x414)) + ((x477 - (x383 * x417)) * Val(2)));
  Val x1049 = ((x1048 + ((x478 - (x386 * x420)) * Val(4))) + ((x479 - (x389 * x423)) * Val(8)));
  Val x1050 = ((x1049 + ((x480 - (x392 * x426)) * Val(16))) + ((x481 - (x395 * x429)) * Val(32)));
  Val x1051 = ((x1050 + ((x482 - (x398 * x432)) * Val(64))) + ((x483 - (x401 * x435)) * Val(128)));
  Val x1052 = ((x1051 + ((x484 - (x404 * x438)) * Val(256))) + ((x485 - (x407 * x441)) * Val(512)));
  Val x1053 =
      ((x1052 + ((x486 - (x410 * x444)) * Val(1024))) + ((x487 - (x413 * x447)) * Val(2048)));
  Val x1054 =
      ((x1053 + ((x488 - (x416 * x450)) * Val(4096))) + ((x489 - (x419 * x453)) * Val(8192)));
  Val x1055 =
      ((x1054 + ((x490 - (x422 * x456)) * Val(16384))) + ((x491 - (x425 * x459)) * Val(32768)));
  // ComputeAE(zirgen/circuit/keccak2/sha2.zir:93)
  Val x1056 = (arg0.a[3][1] * Val(2));
  Val x1057 = (arg0.a[3][2] * Val(4));
  Val x1058 = (arg0.a[3][3] * Val(8));
  Val x1059 = (arg0.a[3][4] * Val(16));
  Val x1060 = (arg0.a[3][5] * Val(32));
  Val x1061 = (arg0.a[3][6] * Val(64));
  Val x1062 = (arg0.a[3][7] * Val(128));
  Val x1063 = (arg0.a[3][8] * Val(256));
  Val x1064 = (arg0.a[3][9] * Val(512));
  Val x1065 = (arg0.a[3][10] * Val(1024));
  Val x1066 = (arg0.a[3][11] * Val(2048));
  Val x1067 = (arg0.a[3][12] * Val(4096));
  Val x1068 = (arg0.a[3][13] * Val(8192));
  Val x1069 = (arg0.a[3][14] * Val(16384));
  Val x1070 = (arg0.a[3][15] * Val(32768));
  Val x1071 = (arg0.a[3][0] + x1056);
  Val x1072 = (((x1071 + x1057) + x1058) + x1059);
  Val x1073 = (((x1072 + x1060) + x1061) + x1062);
  Val x1074 = (((x1073 + x1063) + x1064) + x1065);
  Val x1075 = (((x1074 + x1066) + x1067) + x1068);
  Val x1076 = (arg0.a[3][17] * Val(2));
  Val x1077 = (arg0.a[3][18] * Val(4));
  Val x1078 = (arg0.a[3][19] * Val(8));
  Val x1079 = (arg0.a[3][20] * Val(16));
  Val x1080 = (arg0.a[3][21] * Val(32));
  Val x1081 = (arg0.a[3][22] * Val(64));
  Val x1082 = (arg0.a[3][23] * Val(128));
  Val x1083 = (arg0.a[3][24] * Val(256));
  Val x1084 = (arg0.a[3][25] * Val(512));
  Val x1085 = (arg0.a[3][26] * Val(1024));
  Val x1086 = (arg0.a[3][27] * Val(2048));
  Val x1087 = (arg0.a[3][28] * Val(4096));
  Val x1088 = (arg0.a[3][29] * Val(8192));
  Val x1089 = (arg0.a[3][30] * Val(16384));
  Val x1090 = (arg0.a[3][31] * Val(32768));
  Val x1091 = (arg0.a[3][16] + x1076);
  Val x1092 = (((x1091 + x1077) + x1078) + x1079);
  Val x1093 = (((x1092 + x1080) + x1081) + x1082);
  Val x1094 = (((x1093 + x1083) + x1084) + x1085);
  Val x1095 = (((x1094 + x1086) + x1087) + x1088);
  // DoShaStep(zirgen/circuit/keccak2/top.zir:140)
  UnpackReg_32__16_Struct x1096 =
      exec_CarryAndExpand(ctx,
                          Val2Array{(x798 + (x1031 + x1047)), (x799 + (x1039 + x1055))},
                          LAYOUT_LOOKUP(layout4, a));
  // DoShaStep(zirgen/circuit/keccak2/top.zir:141)
  UnpackReg_32__16_Struct x1097 = exec_CarryAndExpand(
      ctx,
      Val2Array{(x798 + ((x1075 + x1069) + x1070)), (x799 + ((x1095 + x1089) + x1090))},
      LAYOUT_LOOKUP(layout4, e));
  // DoShaStep(zirgen/circuit/keccak2/top.zir:140)
  Val32Array x1098 =
      Val32Array{x1096._super[0]._super._super._super,  x1096._super[1]._super._super._super,
                 x1096._super[2]._super._super._super,  x1096._super[3]._super._super._super,
                 x1096._super[4]._super._super._super,  x1096._super[5]._super._super._super,
                 x1096._super[6]._super._super._super,  x1096._super[7]._super._super._super,
                 x1096._super[8]._super._super._super,  x1096._super[9]._super._super._super,
                 x1096._super[10]._super._super._super, x1096._super[11]._super._super._super,
                 x1096._super[12]._super._super._super, x1096._super[13]._super._super._super,
                 x1096._super[14]._super._super._super, x1096._super[15]._super._super._super,
                 x1096._super[16]._super._super._super, x1096._super[17]._super._super._super,
                 x1096._super[18]._super._super._super, x1096._super[19]._super._super._super,
                 x1096._super[20]._super._super._super, x1096._super[21]._super._super._super,
                 x1096._super[22]._super._super._super, x1096._super[23]._super._super._super,
                 x1096._super[24]._super._super._super, x1096._super[25]._super._super._super,
                 x1096._super[26]._super._super._super, x1096._super[27]._super._super._super,
                 x1096._super[28]._super._super._super, x1096._super[29]._super._super._super,
                 x1096._super[30]._super._super._super, x1096._super[31]._super._super._super};
  // DoShaStep(zirgen/circuit/keccak2/top.zir:141)
  Val32Array x1099 =
      Val32Array{x1097._super[0]._super._super._super,  x1097._super[1]._super._super._super,
                 x1097._super[2]._super._super._super,  x1097._super[3]._super._super._super,
                 x1097._super[4]._super._super._super,  x1097._super[5]._super._super._super,
                 x1097._super[6]._super._super._super,  x1097._super[7]._super._super._super,
                 x1097._super[8]._super._super._super,  x1097._super[9]._super._super._super,
                 x1097._super[10]._super._super._super, x1097._super[11]._super._super._super,
                 x1097._super[12]._super._super._super, x1097._super[13]._super._super._super,
                 x1097._super[14]._super._super._super, x1097._super[15]._super._super._super,
                 x1097._super[16]._super._super._super, x1097._super[17]._super._super._super,
                 x1097._super[18]._super._super._super, x1097._super[19]._super._super._super,
                 x1097._super[20]._super._super._super, x1097._super[21]._super._super._super,
                 x1097._super[22]._super._super._super, x1097._super[23]._super._super._super,
                 x1097._super[24]._super._super._super, x1097._super[25]._super._super._super,
                 x1097._super[26]._super._super._super, x1097._super[27]._super._super._super,
                 x1097._super[28]._super._super._super, x1097._super[29]._super._super._super,
                 x1097._super[30]._super._super._super, x1097._super[31]._super._super._super};
  // DoShaStep(zirgen/circuit/keccak2/top.zir:138)
  NondetBitRegStruct32Array x1100 =
      map(x355,
          LAYOUT_LOOKUP(layout4, w._super),
          ([&](ComponentStruct32Array::value_type x1101,
               BoundLayout<NondetRegLayout32LayoutArray::value_type> x1102) {
            NondetBitRegStruct x1103 = back_NondetBitReg(ctx, 0, x1102);
            return x1103;
          }));
  Val32Array x1104 =
      Val32Array{x1100[0]._super._super,  x1100[1]._super._super,  x1100[2]._super._super,
                 x1100[3]._super._super,  x1100[4]._super._super,  x1100[5]._super._super,
                 x1100[6]._super._super,  x1100[7]._super._super,  x1100[8]._super._super,
                 x1100[9]._super._super,  x1100[10]._super._super, x1100[11]._super._super,
                 x1100[12]._super._super, x1100[13]._super._super, x1100[14]._super._super,
                 x1100[15]._super._super, x1100[16]._super._super, x1100[17]._super._super,
                 x1100[18]._super._super, x1100[19]._super._super, x1100[20]._super._super,
                 x1100[21]._super._super, x1100[22]._super._super, x1100[23]._super._super,
                 x1100[24]._super._super, x1100[25]._super._super, x1100[26]._super._super,
                 x1100[27]._super._super, x1100[28]._super._super, x1100[29]._super._super,
                 x1100[30]._super._super, x1100[31]._super._super};
  return DoShaStepStruct{
      .w = x356,
      .a = x1096,
      .e = x1097,
      .newState = ShaStateStruct{.a = Val32Array4Array{x1098, arg0.a[0], arg0.a[1], arg0.a[2]},
                                 .e = Val32Array4Array{x1099, arg0.e[0], arg0.e[1], arg0.e[2]},
                                 .w = Val32Array16Array{x1104,
                                                        arg0.w[0],
                                                        arg0.w[1],
                                                        arg0.w[2],
                                                        arg0.w[3],
                                                        arg0.w[4],
                                                        arg0.w[5],
                                                        arg0.w[6],
                                                        arg0.w[7],
                                                        arg0.w[8],
                                                        arg0.w[9],
                                                        arg0.w[10],
                                                        arg0.w[11],
                                                        arg0.w[12],
                                                        arg0.w[13],
                                                        arg0.w[14]}}};
}
__device__ ControlStateStruct exec_ShaNextRound(ExecContext& ctx,
                                                ControlStateStruct arg0,
                                                BoundLayout<ShaNextRoundLayout> layout1) {
  // ShaNextRound(zirgen/circuit/keccak2/top.zir:417)
  Val x2 = (arg0.round._super - Val(7));
  NondetRegStruct x3 = exec_IsZero(ctx, x2, LAYOUT_LOOKUP(layout1, isLast));
  // ShaNextRound(zirgen/circuit/keccak2/top.zir:421)
  Val x4 = (arg0.round._super + Val(1));
  ControlStateStruct x5;
  if (to_size_t(x3._super)) {
    // ShaNextRound(zirgen/circuit/keccak2/top.zir:419)
    ControlStateStruct x6 = exec_ControlState(ctx,
                                              Val(10),
                                              arg0.subType._super,
                                              arg0.block._super,
                                              Val(0),
                                              LAYOUT_LOOKUP(layout1, _super.arm0));
    x5 = x6;
  } else if (to_size_t((Val(1) - x3._super))) {
    // ShaNextRound(zirgen/circuit/keccak2/top.zir:421)
    ControlStateStruct x7 = exec_ControlState(ctx,
                                              Val(9),
                                              arg0.subType._super,
                                              arg0.block._super,
                                              x4,
                                              LAYOUT_LOOKUP(layout1, _super.arm1));
    x5 = x7;
  } else {
    assert(0 && "Reached unreachable mux arm");
  }
  // ShaNextRound(zirgen/circuit/keccak2/top.zir:418)
  ControlStateStruct x8 = back_ControlState(ctx, 0, LAYOUT_LOOKUP(layout1, _super._super));
  return x8;
>>>>>>> origin/main
}

} // namespace risc0::circuit::keccak::cuda
