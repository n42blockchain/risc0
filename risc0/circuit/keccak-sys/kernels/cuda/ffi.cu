#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "buffers.h"
#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "witgen.h"
#include "steps.cuh"

#include <cstdint>
#include <cstdio>
#include <cuda/std/array>
#include <string.h>

using namespace risc0;

namespace risc0::circuit::keccak::cuda {

struct ExecBuffers {
  Buffer global;
  Buffer data;
};

struct ScatterInfo {
  uint32_t offset;
  uint32_t row;
  uint16_t col;
  uint16_t count;
  uint32_t bits;
};

__device__ void
nextStep(Buffer* bufData, Buffer* bufGlobal, PreflightTrace* preflight, uint32_t cycle) {
  // printf("nextStep: %u\n", cycle);
  ExecContext ctx(*preflight, cycle);
  MutableBufObj data(ctx, *bufData);
  GlobalBufObj global(ctx, *bufGlobal);
  step_Top(ctx, &data, &global);
}

__global__ void
par_stepExec(Buffer* data, Buffer* global, PreflightTrace* preflight, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }
  nextStep(data, global, preflight, cycle);
}

__global__ void
rev_stepExec(Buffer* data, Buffer* global, PreflightTrace* preflight, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == count - 1) {
    for (uint32_t i = 0; i < count; i++) {
      nextStep(data, global, preflight, count - i - 1);
    }
  }
}

__global__ void
fwd_stepExec(Buffer* data, Buffer* global, PreflightTrace* preflight, uint32_t count) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == 0) {
    for (uint32_t i = 0; i < count; i++) {
      nextStep(data, global, preflight, i);
    }
  }
}

__global__ void scatter_preflight(Fp* into,
                                  const ScatterInfo* infos,
                                  const uint32_t* from,
                                  const uint32_t rows,
                                  const uint32_t count) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count) {
    return;
  }

  const ScatterInfo& info = infos[gid];
  uint32_t innerCount = 32 / info.bits;
  uint32_t mask = (1 << (info.bits)) - 1;
  for (size_t i = 0; i < info.count; i++) {
    uint32_t word = from[info.offset + (i / innerCount)];
    size_t j = i % innerCount;
    uint32_t val = (word >> (j * info.bits)) & mask;
    size_t col = info.col + i;
    into[col * rows + info.row] = val;
  }
}

} // namespace risc0::circuit::keccak::cuda

constexpr size_t kStepModeSeqParallel = 0;
constexpr size_t kStepModeSeqForward = 1;
constexpr size_t kStepModeSeqReverse = 2;

extern "C" {

using namespace risc0::circuit::keccak::cuda;

const char* risc0_circuit_keccak_cuda_witgen(uint32_t mode,
                                             ExecBuffers* buffers,
                                             PreflightTrace* preflight,
                                             uint32_t lastCycle) {
  try {
    Buffer* d_data;
    CUDA_OK(hipMallocManaged(&d_data, sizeof(Buffer)));
    d_data->buf = buffers->data.buf;
    d_data->rows = buffers->data.rows;
    d_data->cols = buffers->data.cols;
    d_data->checkedReads = buffers->data.checkedReads;

    Buffer* d_global;
    CUDA_OK(hipMallocManaged(&d_global, sizeof(Buffer)));
    d_global->buf = buffers->global.buf;
    d_global->rows = buffers->global.rows;
    d_global->cols = buffers->global.cols;
    d_global->checkedReads = buffers->global.checkedReads;

    PreflightTrace* d_preflight;
    CUDA_OK(hipMallocManaged(&d_preflight, sizeof(PreflightTrace)));

    CUDA_OK(hipMalloc(&d_preflight->preimages, preflight->preimagesSize * sizeof(KeccakState)));
    CUDA_OK(hipMemcpy(d_preflight->preimages,
                       preflight->preimages,
                       preflight->preimagesSize * sizeof(KeccakState),
                       hipMemcpyHostToDevice));

    d_preflight->preimagesSize = preflight->preimagesSize;

    CUDA_OK(hipMalloc(&d_preflight->curPreimage, lastCycle * sizeof(uint32_t)));
    CUDA_OK(hipMemcpy(d_preflight->curPreimage,
                       preflight->curPreimage,
                       lastCycle * sizeof(uint32_t),
                       hipMemcpyHostToDevice));

    CudaStream stream;
    auto cfg = getSimpleConfig(lastCycle);
    switch (mode) {
    case kStepModeSeqParallel:
      par_stepExec<<<cfg.grid, cfg.block, 0, stream>>>(d_data, d_global, d_preflight, lastCycle);
      break;
    case kStepModeSeqForward:
      fwd_stepExec<<<cfg.grid, cfg.block, 0, stream>>>(d_data, d_global, d_preflight, lastCycle);
      break;
    case kStepModeSeqReverse:
      rev_stepExec<<<cfg.grid, cfg.block, 0, stream>>>(d_data, d_global, d_preflight, lastCycle);
      break;
    }
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

const char* risc0_circuit_keccak_cuda_scatter(Fp* into,
                                              const ScatterInfo* infos,
                                              const uint32_t* from,
                                              const uint32_t rows,
                                              const uint32_t count) {
  try {
    ScatterInfo* d_infos;
    CUDA_OK(hipMalloc(&d_infos, count * sizeof(ScatterInfo)));
    CUDA_OK(hipMemcpy(d_infos, infos, count * sizeof(ScatterInfo), hipMemcpyHostToDevice));

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    scatter_preflight<<<cfg.grid, cfg.block, 0, stream>>>(into, d_infos, from, rows, count);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

} // extern "C"
